#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/util/cuda_half_util.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

template<typename T>
struct GeluGradFunctor {
  const T coef = std::sqrt(static_cast<T>(2.0) / std::acos(static_cast<T>(-1.0)));
  OF_DEVICE_FUNC T operator()(T x, T dy) const {
    return static_cast<T>(0.5)
           * (static_cast<T>(1.0) + erf(static_cast<T>(M_SQRT1_2) * x)
              + x * coef * exp(static_cast<T>(-0.5) * x * x))
           * dy;
  }
};

template<>
struct GeluGradFunctor<half> {
  GeluGradFunctor<float> float_functor;
  OF_DEVICE_FUNC half operator()(half x, half dy) const {
    return __float2half(float_functor(__half2float(x), __half2float(dy)));
  }
};

}  // namespace

template<typename T>
class GpuGeluGradKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  GpuGeluGradKernel() = default;
  ~GpuGeluGradKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const int64_t elem_cnt = x->shape().elem_cnt();
    OF_CUDA_CHECK((cuda::elementwise::Binary(GeluGradFunctor<T>(), elem_cnt, dx->mut_dptr<T>(),
                                             x->dptr<T>(), dy->dptr<T>(),
                                             ctx->stream()->As<ep::CudaStream>()->hip_stream())));
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_GELU_GRAD_KERNEL(dtype)                          \
  REGISTER_USER_KERNEL("gelu_grad")                                    \
      .SetCreateFn<GpuGeluGradKernel<dtype>>()                         \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_CUDA_GELU_GRAD_KERNEL(float)
REGISTER_CUDA_GELU_GRAD_KERNEL(double)
REGISTER_CUDA_GELU_GRAD_KERNEL(half)

}  // namespace oneflow
