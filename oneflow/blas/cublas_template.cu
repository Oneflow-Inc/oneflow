#include "oneflow/blas/cublas_template.h"

namespace oneflow {

template<>
void cublas_gemm<float>(
    const hipblasHandle_t& cublas_handle, const hipblasOperation_t cuTransA,
    const hipblasOperation_t cuTransB, const int M, const int N, const int K,
    const float* alpha, const float* A, const int lda, const float* B,
    const int ldb, const float* beta, float* C, const int ldc) {
  CHECK_EQ(hipblasSgemm(
               cublas_handle, cuTransA, cuTransB, M, N, K, alpha, A, lda, B,
               ldb, beta, C, ldc),
           HIPBLAS_STATUS_SUCCESS);
}

template<>
void cublas_gemm<double>(
    const hipblasHandle_t& cublas_handle, const hipblasOperation_t cuTransA,
    const hipblasOperation_t cuTransB, const int M, const int N, const int K,
    const double* alpha, const double* A, const int lda, const double* B,
    const int ldb, const double* beta, double* C, const int ldc) {
  CHECK_EQ(hipblasDgemm(
               cublas_handle, cuTransA, cuTransB, M, N, K, alpha, A, lda, B,
               ldb, beta, C, ldc),
           HIPBLAS_STATUS_SUCCESS);
}

}  // namespace oneflow
