//#ifdef USE_CUDNN
#include <vector>

#include "layers/cudnn_pooling_layer.h"
#include "layers/layer_factory.h"

namespace caffe {

template <typename Dtype>
void CuDNNPoolingLayer<Dtype>::Forward(const ContextParam& ctx,
  DataParam<Dtype>* data_param, ModelParam<Dtype>* model_param) const {

  GET_CONCRETE_POINTER(CuDNNPoolingData, data, data_param);
  GET_CONCRETE_POINTER(CuDNNPoolingParam, param, param_);

  CHECK(ctx.hip_stream) << "Default stream is not allowed";
  CUDNN_CHECK(hipdnnSetStream(ctx.cudnn_handle, ctx.hip_stream));

  CHECK_NOTNULL(data->in);
  CHECK_NOTNULL(data->out);
  // CHECK_NOTNULL(data->idx);
  const Dtype* in_data = data->in->data();
  Dtype* out_data = data->out->mutable_data();
  int count = data->out->shape().count();

  CUDNN_CHECK(hipdnnPoolingForward(ctx.cudnn_handle, param->pooling_desc_,
    cudnn::dataType<Dtype>::one,
    param->in_desc_, in_data,
    cudnn::dataType<Dtype>::zero,
    param->out_desc_, out_data));

  // if (ctx.hip_stream) {
  //   CUDA_CHECK(hipMemcpyAsync(data->idx->mutable_data(),
  //     data->out->data(), count*sizeof(Dtype),
  //     hipMemcpyDeviceToDevice, ctx.hip_stream));
  // } else {
  //   CUDA_CHECK(hipMemcpy(data->idx->mutable_data(),
  //     data->out->data(), count*sizeof(Dtype),
  //     hipMemcpyDeviceToDevice));
  // }
}

template <typename Dtype>
void CuDNNPoolingLayer<Dtype>::Backward(const ContextParam& ctx,
  DataParam<Dtype>* data_param, ModelParam<Dtype>* model_param) const {

  GET_CONCRETE_POINTER(CuDNNPoolingData, data, data_param);
  GET_CONCRETE_POINTER(CuDNNPoolingParam, param, param_);

  CHECK(ctx.hip_stream) << "Default stream is not allowed";
  CUDNN_CHECK(hipdnnSetStream(ctx.cudnn_handle, ctx.hip_stream));

  // Use ctx, data and model
  CHECK_NOTNULL(data->in);
  CHECK_NOTNULL(data->out);
  CHECK_NOTNULL(data->in_diff);
  CHECK_NOTNULL(data->out_diff);
  // CHECK_NOTNULL(data->idx);
  const Dtype* out_diff_ = data->out_diff->data();
  Dtype* in_diff_ = data->in_diff->mutable_data();

  // Dtype* inputs_gpu_data;
  // CUDA_CHECK(hipMalloc(&inputs_gpu_data, data->in->shape().count()*sizeof(Dtype)));
  // CUDA_CHECK(hipMemcpy(inputs_gpu_data, data->in->data(),
  //   data->in->shape().count() * sizeof(Dtype),
  //   hipMemcpyDeviceToDevice));

  CUDNN_CHECK(hipdnnPoolingBackward(ctx.cudnn_handle, param->pooling_desc_,
    cudnn::dataType<Dtype>::one,
    param->out_desc_, data->out->data(),
    param->out_desc_, out_diff_,
    param->in_desc_, data->in->data(),
    cudnn::dataType<Dtype>::zero,
    param->in_desc_, in_diff_));
}

INSTANTIATE_LAYER_FUNCS(CuDNNPoolingLayer);

}  // namespace caffe
//#endif