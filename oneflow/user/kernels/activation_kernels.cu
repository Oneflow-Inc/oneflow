#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/activation_kernels.h"
#include "oneflow/user/kernels/elementwise_xpu_kernel.cuh"

namespace oneflow {

template<>
struct EluFunctor<half> {
  OF_DEVICE_FUNC explicit EluFunctor(float alpha)
      : alpha(alpha), float_functor(EluFunctor<float>(alpha)) {}
  OF_DEVICE_FUNC half operator()(half x) const {
    return __float2half(float_functor(__half2float(x)));
  }
  const float alpha;
  EluFunctor<float> float_functor;
};

template<>
struct EluGradFunctor<half> {
  OF_DEVICE_FUNC explicit EluGradFunctor(float alpha)
      : alpha(alpha), float_functor(EluGradFunctor<float>(alpha)) {}
  OF_DEVICE_FUNC half operator()(half x, half dy) const {
    return __float2half(float_functor(__half2float(x), __half2float(dy)));
  }
  const float alpha;
  EluGradFunctor<float> float_functor;
};

template<>
struct LeakyReluFunctor<half> {
  OF_DEVICE_FUNC explicit LeakyReluFunctor(float alpha) : alpha(alpha) {}
  __device__ half operator()(half x) const {
    half zero = __float2half(0);
    return (x > zero) ? x : __float2half(alpha) * x;
  }
  const float alpha;
};

template<>
struct LeakyReluGradFunctor<half> {
  OF_DEVICE_FUNC explicit LeakyReluGradFunctor(float alpha) : alpha(alpha) {}
  __device__ half operator()(half x, half dy) const {
    half zero = __float2half(0);
    return (x > zero) ? dy : __float2half(alpha) * dy;
  }
  const float alpha;
};

template<>
struct SoftplusFunctor<half> {
  OF_DEVICE_FUNC explicit SoftplusFunctor(float beta, float threshold)
      : beta(beta), threshold(threshold), float_functor(SoftplusFunctor<float>(beta, threshold)) {}
  __device__ half operator()(half x) const { return __float2half(float_functor(__half2float(x))); }
  const float beta;
  const float threshold;
  SoftplusFunctor<float> float_functor;
};

template<>
struct SoftplusGradFunctor<half> {
  OF_DEVICE_FUNC explicit SoftplusGradFunctor(float beta, float threshold)
      : beta(beta),
        threshold(threshold),
        float_functor(SoftplusGradFunctor<float>(beta, threshold)) {}
  __device__ half operator()(half x, half dy) const {
    return __float2half(float_functor(__half2float(x), __half2float(dy)));
  }
  const float beta;
  const float threshold;
  SoftplusGradFunctor<float> float_functor;
};

template<>
struct CeluFunctor<half> {
  OF_DEVICE_FUNC explicit CeluFunctor(float alpha)
      : alpha(alpha), float_functor(CeluFunctor<float>(alpha)) {}
  OF_DEVICE_FUNC half operator()(half x) const {
    return __float2half(float_functor(__half2float(x)));
  }
  const float alpha;
  CeluFunctor<float> float_functor;
};

template<>
struct CeluGradFunctor<half> {
  OF_DEVICE_FUNC explicit CeluGradFunctor(float alpha)
      : alpha(alpha), float_functor(CeluGradFunctor<float>(alpha)) {}
  OF_DEVICE_FUNC half operator()(half x, half dy) const {
    return __float2half(float_functor(__half2float(x), __half2float(dy)));
  }
  const float alpha;
  CeluGradFunctor<float> float_functor;
};

template<>
struct HardswishFunctor<half> {
  HardswishFunctor<float> float_functor;
  OF_DEVICE_FUNC half operator()(half x) const {
    return __float2half(float_functor(__half2float(x)));
  }
};

template<>
struct HardswishGradFunctor<half> {
  HardswishGradFunctor<float> float_functor;
  OF_DEVICE_FUNC half operator()(half x, half dy) const {
    return __float2half(float_functor(__half2float(x), __half2float(dy)));
  }
};

template<>
struct MishFunctor<half> {
  OF_DEVICE_FUNC explicit MishFunctor() : float_functor(MishFunctor<float>()) {}
  OF_DEVICE_FUNC half operator()(half x) const {
    return __float2half(float_functor(__half2float(x)));
  }
  MishFunctor<float> float_functor;
};

template<>
struct MishGradFunctor<half> {
  OF_DEVICE_FUNC explicit MishGradFunctor() : float_functor(MishGradFunctor<float>()) {}
  OF_DEVICE_FUNC half operator()(half x, half dy) const {
    return __float2half(float_functor(__half2float(x), __half2float(dy)));
  }
  MishGradFunctor<float> float_functor;
};

template<>
struct SiluFunctor<half> {
  OF_DEVICE_FUNC explicit SiluFunctor() : float_functor(SiluFunctor<float>()) {}
  OF_DEVICE_FUNC half operator()(half x) const {
    return __float2half(float_functor(__half2float(x)));
  }
  SiluFunctor<float> float_functor;
};

template<>
struct SiluGradFunctor<half> {
  OF_DEVICE_FUNC explicit SiluGradFunctor() : float_functor(SiluGradFunctor<float>()) {}
  OF_DEVICE_FUNC half operator()(half x, half dy) const {
    return __float2half(float_functor(__half2float(x), __half2float(dy)));
  }
  SiluGradFunctor<float> float_functor;
};

template<>
struct SeluFunctor<half> {
  OF_DEVICE_FUNC explicit SeluFunctor() : float_functor(SeluFunctor<float>()) {}
  OF_DEVICE_FUNC half operator()(half x) const {
    return __float2half(float_functor(__half2float(x)));
  }
  SeluFunctor<float> float_functor;
};

template<>
struct SeluGradFunctor<half> {
  OF_DEVICE_FUNC explicit SeluGradFunctor() : float_functor(SeluGradFunctor<float>()) {}
  OF_DEVICE_FUNC half operator()(half x, half dy) const {
    return __float2half(float_functor(__half2float(x), __half2float(dy)));
  }
  SeluGradFunctor<float> float_functor;
};

template<>
struct SoftSignFunctor<half> {
  OF_DEVICE_FUNC explicit SoftSignFunctor() : float_functor(SoftSignFunctor<float>()) {}
  OF_DEVICE_FUNC half operator()(half x) const {
    return __float2half(float_functor(__half2float(x)));
  }
  SoftSignFunctor<float> float_functor;
};

template<>
struct SoftSignGradFunctor<half> {
  OF_DEVICE_FUNC explicit SoftSignGradFunctor() : float_functor(SoftSignGradFunctor<float>()) {}
  OF_DEVICE_FUNC half operator()(half x, half dy) const {
    return __float2half(float_functor(__half2float(x), __half2float(dy)));
  }
  SoftSignGradFunctor<float> float_functor;
};

template<>
struct ReluGradFunctor<half> {
  OF_DEVICE_FUNC explicit ReluGradFunctor() {}
  __device__ half operator()(half y, half dy) const {
    half zero = __float2half(0.0);
    if (__hgt(y, zero)) {
      return dy;
    } else {
      return zero;
    }
  }
};

template<>
struct SoftShrinkFunctor<half> {
  OF_DEVICE_FUNC explicit SoftShrinkFunctor(float alpha)
      : alpha(alpha), float_functor(SoftShrinkFunctor<float>(alpha)) {}
  OF_DEVICE_FUNC half operator()(half x) const {
    return __float2half(float_functor(__half2float(x)));
  }
  const float alpha;
  SoftShrinkFunctor<float> float_functor;
};

template<>
struct SoftShrinkGradFunctor<half> {
  OF_DEVICE_FUNC explicit SoftShrinkGradFunctor(float alpha)
      : alpha(alpha), float_functor(SoftShrinkGradFunctor<float>(alpha)) {}
  OF_DEVICE_FUNC half operator()(half y, half dy) const {
    return __float2half(float_functor(__half2float(y), __half2float(dy)));
  }

  const float alpha;
  SoftShrinkGradFunctor<float> float_functor;
};

#define REGISTER_ACTIVATION_CUDA_KERNEL(dtype)           \
  REGISTER_ELU_KERNEL(DeviceType::kCUDA, dtype);         \
  REGISTER_CELU_KERNEL(DeviceType::kCUDA, dtype);        \
  REGISTER_HARDSWISH_KERNEL(DeviceType::kCUDA, dtype);   \
  REGISTER_HARDSIGMOID_KERNEL(DeviceType::kCUDA, dtype); \
  REGISTER_HARDTANH_KERNEL(DeviceType::kCUDA, dtype);    \
  REGISTER_MISH_KERNEL(DeviceType::kCUDA, dtype);        \
  REGISTER_SILU_KERNEL(DeviceType::kCUDA, dtype);        \
  REGISTER_SELU_KERNEL(DeviceType::kCUDA, dtype);        \
  REGISTER_SOFTSHRINK_KERNEL(DeviceType::kCUDA, dtype);  \
  REGISTER_SOFTSIGN_KERNEL(DeviceType::kCUDA, dtype);    \
  REGISTER_LEAKYRELU_KERNEL(DeviceType::kCUDA, dtype);   \
  REGISTER_SOFTPLUS_KERNEL(DeviceType::kCUDA, dtype);    \
  REGISTER_RELU_BACKWARD_KERNEL(DeviceType::kCUDA, dtype);

namespace {

REGISTER_ACTIVATION_CUDA_KERNEL(half);
REGISTER_ACTIVATION_CUDA_KERNEL(float);
REGISTER_ACTIVATION_CUDA_KERNEL(double);

}  // namespace

}  // namespace oneflow
