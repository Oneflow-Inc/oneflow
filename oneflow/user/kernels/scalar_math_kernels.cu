#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/scalar_math_kernels.h"
#include "oneflow/user/kernels/elementwise_xpu_kernel.cuh"
#include "oneflow/core/kernel/util/cuda_half_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

template<template<typename> class Op, typename T>
struct UnaryByScalarFunctor {
  __host__ __device__ explicit UnaryByScalarFunctor(T scalar) : scalar(scalar) {}
  __device__ T operator()(T a) const { return Op<T>::Invoke(a, scalar); }
  const T scalar;
};

template<template<typename> class Op, typename T>
struct UnaryByScalarReverseFunctor {
  __host__ __device__ explicit UnaryByScalarReverseFunctor(T scalar) : scalar(scalar) {}
  __device__ T operator()(T a) const { return Op<T>::Invoke(scalar, a); }
  const T scalar;
};

template<template<typename> class Op>
struct UnaryByScalarFunctor<Op, float16> {
  __host__ __device__ explicit UnaryByScalarFunctor(half scalar) : scalar(scalar) {}
  __device__ half operator()(half a) const { return Op<half>::Invoke(a, scalar); }
  const half scalar;
};

template<template<typename> class Op>
struct UnaryByScalarReverseFunctor<Op, float16> {
  __host__ __device__ explicit UnaryByScalarReverseFunctor(half scalar) : scalar(scalar) {}
  __device__ half operator()(half a) const { return Op<half>::Invoke(scalar, a); }
  const half scalar;
};

template<template<typename> class BIN_OP, typename T>
struct ScalarMathFunctor<DeviceType::kCUDA, BIN_OP, T> final {
  void operator()(ep::Stream* stream, const int64_t elem_cnt, const T scalar, const T* in, T* out) {
    OF_CUDA_CHECK(cuda::elementwise::Unary(UnaryByScalarFunctor<BIN_OP, T>(scalar), elem_cnt, out,
                                           in, stream->As<ep::CudaStream>()->hip_stream()));
  }
};

template<template<typename> class BIN_OP>
struct ScalarMathFunctor<DeviceType::kCUDA, BIN_OP, float16> final {
  void operator()(ep::Stream* stream, const int64_t elem_cnt, float16 scalar, const float16* in,
                  float16* out) {
    OF_CUDA_CHECK(cuda::elementwise::Unary(
        UnaryByScalarFunctor<BIN_OP, float16>(float16_2half(scalar)), elem_cnt,
        reinterpret_cast<half*>(out), reinterpret_cast<const half*>(in),
        stream->As<ep::CudaStream>()->hip_stream()));
  }
};

template<template<typename> class BIN_OP, typename T>
struct ScalarReverseMathFunctor<DeviceType::kCUDA, BIN_OP, T> final {
  void operator()(ep::Stream* stream, const int64_t elem_cnt, const T scalar, const T* in, T* out) {
    OF_CUDA_CHECK(cuda::elementwise::Unary(UnaryByScalarReverseFunctor<BIN_OP, T>(scalar), elem_cnt,
                                           out, in, stream->As<ep::CudaStream>()->hip_stream()));
  }
};

template<template<typename> class BIN_OP>
struct ScalarReverseMathFunctor<DeviceType::kCUDA, BIN_OP, float16> final {
  void operator()(ep::Stream* stream, const int64_t elem_cnt, float16 scalar, const float16* in,
                  float16* out) {
    OF_CUDA_CHECK(cuda::elementwise::Unary(
        UnaryByScalarReverseFunctor<BIN_OP, float16>(float16_2half(scalar)), elem_cnt,
        reinterpret_cast<half*>(out), reinterpret_cast<const half*>(in),
        stream->As<ep::CudaStream>()->hip_stream()));
  }
};

INSTANTIATE_SCALAR_MATH_FUNCTORS(DeviceType::kCUDA, BinaryFuncAdd);
INSTANTIATE_SCALAR_MATH_FUNCTORS(DeviceType::kCUDA, BinaryFuncFloorDiv);
INSTANTIATE_SCALAR_MATH_FUNCTORS(DeviceType::kCUDA, BinaryFuncFMod);
INSTANTIATE_SCALAR_MATH_FUNCTORS(DeviceType::kCUDA, BinaryFuncMul);
INSTANTIATE_SCALAR_MATH_FUNCTORS(DeviceType::kCUDA, BinaryFuncDiv);
INSTANTIATE_SCALAR_MATH_FUNCTORS(DeviceType::kCUDA, BinaryFuncPow);
INSTANTIATE_SCALAR_REVERSE_MATH_FUNCTORS(DeviceType::kCUDA, BinaryFuncPow);

template<typename T>
struct ScalarPowGradFunctor {
  OF_DEVICE_FUNC explicit ScalarPowGradFunctor(T exponent) : exponent(exponent) {}
  __device__ T operator()(T x, T dy) const {
    return exponent * (pow(x, exponent - static_cast<T>(1.0))) * dy;
  }
  const T exponent;
};

template<>
struct ScalarPowGradFunctor<half> {
  OF_DEVICE_FUNC explicit ScalarPowGradFunctor(half exponent) : exponent(exponent) {}
  __device__ half operator()(half x, half dy) const {
    return __float2half(__half2float(exponent)
                        * (powf(__half2float(x), __half2float(exponent) - static_cast<float>(1.0)))
                        * __half2float(dy));
  }
  const half exponent;
};

template<typename T>
struct ScalarReversePowGradFunctor {
  OF_DEVICE_FUNC explicit ScalarReversePowGradFunctor(T exponent) : exponent(exponent) {}
  __device__ T operator()(T x, T dy) const { return pow(exponent, x) * log(exponent) * dy; }
  const T exponent;
};

template<>
struct ScalarReversePowGradFunctor<float> {
  OF_DEVICE_FUNC explicit ScalarReversePowGradFunctor(float exponent) : exponent(exponent) {}
  __device__ float operator()(float x, float dy) const {
    return powf(exponent, x) * logf(exponent) * dy;
  }
  const float exponent;
};

template<>
struct ScalarReversePowGradFunctor<half> {
  OF_DEVICE_FUNC explicit ScalarReversePowGradFunctor(half exponent) : exponent(exponent) {}
  __device__ half operator()(half x, half dy) const {
    const float exp = __half2float(exponent);
    return __float2half(exp * powf(exp, __half2float(x)) * logf(exp) * __half2float(dy));
  }
  const half exponent;
};

template<DeviceType device_type, typename T>
class GpuScalarPowGradKernel final : public user_op::OpKernel {
 public:
  GpuScalarPowGradKernel() = default;
  ~GpuScalarPowGradKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_tensor = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const T* x_ptr = x_tensor->dptr<T>();
    const T* dy_ptr = dy_tensor->dptr<T>();
    T* dx_ptr = dx_tensor->mut_dptr<T>();
    T scalar_operand = static_cast<T>(0);
    if (ctx->Attr<bool>("has_int_operand")) {
      scalar_operand = static_cast<T>(ctx->Attr<int64_t>("int_operand"));
    } else if (ctx->Attr<bool>("has_float_operand")) {
      scalar_operand = static_cast<T>(ctx->Attr<double>("float_operand"));
    } else {
      UNIMPLEMENTED();
    }
    const int32_t elem_cnt = x_tensor->shape_view().elem_cnt();
    OF_CUDA_CHECK((oneflow::cuda::elementwise::Binary(
        ScalarPowGradFunctor<T>(scalar_operand), elem_cnt, dx_ptr, x_ptr, dy_ptr,
        ctx->stream()->As<ep::CudaStream>()->hip_stream())));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_SCALAR_POW_BACKWARD_KERNEL(device, dtype) \
  REGISTER_USER_KERNEL("scalar_pow_grad")                       \
      .SetCreateFn<GpuScalarPowGradKernel<device, dtype>>()     \
      .SetIsMatchedHob((user_op::HobDeviceType() == device)     \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_CUDA_SCALAR_POW_BACKWARD_KERNEL(DeviceType::kCUDA, float);
REGISTER_CUDA_SCALAR_POW_BACKWARD_KERNEL(DeviceType::kCUDA, double);

template<DeviceType device_type, typename T>
class GpuScalarReversePowGradKernel final : public user_op::OpKernel {
 public:
  GpuScalarReversePowGradKernel() = default;
  ~GpuScalarReversePowGradKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_tensor = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const T* x_ptr = x_tensor->dptr<T>();
    const T* dy_ptr = dy_tensor->dptr<T>();
    T* dx_ptr = dx_tensor->mut_dptr<T>();
    T scalar_operand = static_cast<T>(0);
    if (ctx->Attr<bool>("has_int_operand")) {
      scalar_operand = static_cast<T>(ctx->Attr<int64_t>("int_operand"));
    } else if (ctx->Attr<bool>("has_float_operand")) {
      scalar_operand = static_cast<T>(ctx->Attr<double>("float_operand"));
    } else {
      UNIMPLEMENTED();
    }
    const int32_t elem_cnt = x_tensor->shape_view().elem_cnt();
    OF_CUDA_CHECK((oneflow::cuda::elementwise::Binary(
        ScalarReversePowGradFunctor<T>(scalar_operand), elem_cnt, dx_ptr, x_ptr, dy_ptr,
        ctx->stream()->As<ep::CudaStream>()->hip_stream())));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_SCALAR_REVERSE_POW_BACKWARD_KERNEL(device, dtype) \
  REGISTER_USER_KERNEL("scalar_reverse_pow_grad")                       \
      .SetCreateFn<GpuScalarReversePowGradKernel<device, dtype>>()      \
      .SetIsMatchedHob((user_op::HobDeviceType() == device)             \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_CUDA_SCALAR_REVERSE_POW_BACKWARD_KERNEL(DeviceType::kCUDA, float);
REGISTER_CUDA_SCALAR_REVERSE_POW_BACKWARD_KERNEL(DeviceType::kCUDA, double);

}  // namespace oneflow
