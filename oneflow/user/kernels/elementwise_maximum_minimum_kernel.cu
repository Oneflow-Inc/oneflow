
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#ifdef WITH_CUDA
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/user/kernels/elementwise_maximum_minimum_kernel.h"
#include "oneflow/core/ep/cuda/cuda_stream.h"

namespace oneflow {

namespace {
template<template<typename> class Opt, typename T>
__global__ void ElementwiseXimumGradGpuKernel(int64_t elem_cnt, const T* dz, const T* x, const T* y,
                                              T* dx, T* dy) {
  XPU_1D_KERNEL_LOOP(idx, elem_cnt) {
    Opt<T>()(dz[idx], x[idx], y[idx], dx ? &dx[idx] : nullptr, dy ? &dy[idx] : nullptr);
  }
}

template<template<typename> class Opt, typename T>
struct ElemwiseXimumGradFunctor<DeviceType::kCUDA, Opt, T> final {
  void operator()(ep::Stream* stream, int64_t elem_cnt, const T* dz, const T* x, const T* y, T* dx,
                  T* dy) {
    ElementwiseXimumGradGpuKernel<Opt, T>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
           stream->As<ep::CudaStream>()->cuda_stream()>>>(elem_cnt, dz, x, y, dx, dy);
  }
};

template<template<typename> class Opt, typename T>
struct ElemwiseXimumFunctor<DeviceType::kCUDA, Opt, T> final {
  void operator()(ep::Stream* stream, int64_t elem_cnt, T* z, const T* x, const T* y) {
    OF_CUDA_CHECK(cuda::elementwise::Binary(Opt<T>(), elem_cnt, z, x, y,
                                            stream->As<ep::CudaStream>()->cuda_stream()));
  }
};
}  // namespace

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_MAXIMUM_KERNELS, (DeviceType::kCUDA),
                                 ARITHMETIC_DATA_TYPE_SEQ UNSIGNED_INT_DATA_TYPE_SEQ)
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_MINIMUM_KERNELS, (DeviceType::kCUDA),
                                 ARITHMETIC_DATA_TYPE_SEQ UNSIGNED_INT_DATA_TYPE_SEQ)
}  // namespace oneflow
#endif  // WITH_CUDA
