#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/user/kernels/lerp_kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void LerpForwardGpu(const int n, const T* start, const T* weight, const T* end, T* out) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T start_i = start[i];
    out[i] = start_i + weight[i] * (end[i] - start_i);
  }
}

template<typename T, typename ValueT>
__global__ void ScalarLerpForwardGpu(const int n, const T* start, const ValueT weight, const T* end,
                                     T* out) {
  T weight_calculate = 0.0;
  if constexpr (std::is_same<T, half>::value) {
    weight_calculate = __float2half(static_cast<float>(weight));
  } else {
    weight_calculate = static_cast<T>(weight);
  }
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T start_i = start[i];
    out[i] = start_i + weight_calculate * (end[i] - start_i);
  }
}

template<typename T>
__global__ void LerpBackwardGpu(const int n, const T* start, const T* weight, const T* end,
                                const T* out_diff, T* start_diff, T* weight_diff, T* end_diff) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T out_diff_i = out_diff[i];
    const T start_diff_i = (static_cast<T>(1.0) - weight[i]) * out_diff_i;
    start_diff[i] = start_diff_i;
    weight_diff[i] = (end[i] - start[i]) * out_diff_i;
    end_diff[i] = out_diff_i - start_diff_i;
  }
}

template<typename T, typename ValueT>
__global__ void ScalarLerpBackwardGpu(const int n, const T* start, const ValueT weight,
                                      const T* end, const T* out_diff, T* start_diff, T* end_diff) {
  T weight_calculate = 0.0;
  if constexpr (std::is_same<T, half>::value) {
    weight_calculate = __float2half(static_cast<float>(weight));
  } else {
    weight_calculate = static_cast<T>(weight);
  }
  CUDA_1D_KERNEL_LOOP(i, n) {
    T out_diff_i = out_diff[i];
    const T start_diff_i = (static_cast<T>(1.0) - weight_calculate) * out_diff_i;
    start_diff[i] = start_diff_i;
    end_diff[i] = out_diff_i - start_diff_i;
  }
}

}  // namespace

template<typename T>
struct LerpKernelUtil<DeviceType::kCUDA, T> {
  static void Forward(ep::Stream* stream, const int64_t n, const T* start, const T* weight,
                      const T* end, T* out) {
    RUN_CUDA_KERNEL((LerpForwardGpu<T>), stream, n, n, start, weight, end, out);
  }

  static void Backward(ep::Stream* stream, const int64_t n, const T* start, const T* weight,
                       const T* end, const T* out_diff, T* start_diff, T* weight_diff,
                       T* end_diff) {
    RUN_CUDA_KERNEL((LerpBackwardGpu<T>), stream, n, n, start, weight, end, out_diff, start_diff,
                    weight_diff, end_diff);
  }
};

template<typename T, typename ValueT>
struct ScalarLerpKernelUtil<DeviceType::kCUDA, T, ValueT> {
  static void Forward(ep::Stream* stream, const int64_t n, const T* start, const T* end,
                      const Scalar operand, T* out) {
    ValueT weight = operand.Value<ValueT>();
    RUN_CUDA_KERNEL((ScalarLerpForwardGpu<T, ValueT>), stream, n, n, start, weight, end, out);
  }

  static void Backward(ep::Stream* stream, const int64_t n, const T* start, const T* end,
                       const T* out_diff, const Scalar operand, T* start_diff, T* end_diff) {
    ValueT weight = operand.Value<ValueT>();
    RUN_CUDA_KERNEL((ScalarLerpBackwardGpu<T, ValueT>), stream, n, n, start, weight, end, out_diff,
                    start_diff, end_diff);
  }
};

#define INSTANTIATE_LERP_KERNEL_UTIL_CUDA(data_type, other) \
  template struct LerpKernelUtil<DeviceType::kCUDA, data_type>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_LERP_KERNEL_UTIL_CUDA, LERP_DATA_TYPE_SEQ_CUDA)
#undef INSTANTIATE_LERP_KERNEL_UTIL_CUDA

#define INSTANTIATE_SCALAR_LERP_KERNEL_UTIL_CUDA(data_type, value_data_type)           \
  template struct ScalarLerpKernelUtil<DeviceType::kCUDA, OF_PP_PAIR_FIRST(data_type), \
                                       OF_PP_PAIR_FIRST(value_data_type)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_SCALAR_LERP_KERNEL_UTIL_CUDA, LERP_DATA_TYPE_SEQ_CUDA,
                                 SCALAR_VALUE_DATA_TYPE_SEQ)
#undef INSTANTIATE_SCALAR_LERP_KERNEL_UTIL_CUDA

}  // namespace oneflow
