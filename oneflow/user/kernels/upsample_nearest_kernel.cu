#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/common/nd_index_offset_helper.h"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/user/kernels/upsample_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void UpsampleNearest1DForward(const int64_t elem_cnt, const T* in_dptr,
                                         NdIndexOffsetHelper<int64_t, 3> in_helper,
                                         NdIndexOffsetHelper<int64_t, 3> out_helper,
                                         const int64_t in_height, const float scale_factor,
                                         T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h;
    out_helper.OffsetToNdIndex(index, n, c, h);
    const int64_t in_h = GetNearestInputIndex(h, scale_factor, in_height);
    out_dptr[index] = in_dptr[in_helper.NdIndexToOffset(n, c, in_h)];
  }
}

template<typename T>
__global__ void UpsampleNearest1DBackward(const int64_t elem_cnt, const T* dy_dptr,
                                          NdIndexOffsetHelper<int64_t, 3> dy_helper,
                                          NdIndexOffsetHelper<int64_t, 3> dx_helper,
                                          const int64_t in_height, const float scale_factor,
                                          T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h;
    dy_helper.OffsetToNdIndex(index, n, c, h);
    const int64_t dx_h = GetNearestInputIndex(h, scale_factor, in_height);
    cuda::atomic::Add(dx_dptr + dx_helper.NdIndexToOffset(n, c, dx_h), dy_dptr[index]);
  }
}

template<typename T>
__global__ void UpsampleNearest2DForward(const int64_t elem_cnt, const T* in_dptr,
                                         NdIndexOffsetHelper<int64_t, 4> in_helper,
                                         NdIndexOffsetHelper<int64_t, 4> out_helper,
                                         const int64_t in_height, const int64_t in_width,
                                         const float scale_h, const float scale_w, T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    out_helper.OffsetToNdIndex(index, n, c, h, w);
    const int64_t in_h = GetNearestInputIndex(h, scale_h, in_height);
    const int64_t in_w = GetNearestInputIndex(w, scale_w, in_width);
    out_dptr[index] = in_dptr[in_helper.NdIndexToOffset(n, c, in_h, in_w)];
  }
}

template<typename T>
__global__ void UpsampleNearest2DBackward(const int64_t elem_cnt, const T* dy_dptr,
                                          NdIndexOffsetHelper<int64_t, 4> dy_helper,
                                          NdIndexOffsetHelper<int64_t, 4> dx_helper,
                                          const int64_t dx_height, const int64_t dx_width,
                                          const float scale_h, const float scale_w, T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    dy_helper.OffsetToNdIndex(index, n, c, h, w);
    const int64_t dx_h = GetNearestInputIndex(h, scale_h, dx_height);
    const int64_t dx_w = GetNearestInputIndex(w, scale_w, dx_width);
    cuda::atomic::Add(dx_dptr + dx_helper.NdIndexToOffset(n, c, dx_h, dx_w), dy_dptr[index]);
  }
}

template<typename T>
__global__ void UpsampleNearest3DForward(const int64_t elem_cnt, const T* in_dptr,
                                         NdIndexOffsetHelper<int64_t, 5> in_helper,
                                         NdIndexOffsetHelper<int64_t, 5> out_helper,
                                         const int64_t in_depth, const int64_t in_height,
                                         const int64_t in_width, const float scale_d,
                                         const float scale_h, const float scale_w, T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, d, h, w;
    out_helper.OffsetToNdIndex(index, n, c, d, h, w);
    const int64_t in_h = GetNearestInputIndex(h, scale_h, in_height);
    const int64_t in_w = GetNearestInputIndex(w, scale_w, in_width);
    const int64_t in_d = GetNearestInputIndex(d, scale_d, in_depth);
    out_dptr[index] = in_dptr[in_helper.NdIndexToOffset(n, c, in_d, in_h, in_w)];
  }
}

template<typename T>
__global__ void UpsampleNearest3DBackward(const int64_t elem_cnt, const T* dy_dptr,
                                          NdIndexOffsetHelper<int64_t, 5> dy_helper,
                                          NdIndexOffsetHelper<int64_t, 5> dx_helper,
                                          const int64_t in_depth, const int64_t in_height,
                                          const int64_t in_width, const float scale_d,
                                          const float scale_h, const float scale_w, T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, d, h, w;
    dy_helper.OffsetToNdIndex(index, n, c, d, h, w);
    const int64_t dx_h = GetNearestInputIndex(h, scale_h, in_height);
    const int64_t dx_w = GetNearestInputIndex(w, scale_w, in_width);
    const int64_t in_d = GetNearestInputIndex(d, scale_d, in_depth);
    cuda::atomic::Add(dx_dptr + dx_helper.NdIndexToOffset(n, c, in_d, dx_h, dx_w), dy_dptr[index]);
  }
}

}  // namespace

template<typename T>
class UpsampleNearest1DGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearest1DGPUKernel() = default;
  ~UpsampleNearest1DGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_tensor = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_tensor = ctx->Tensor4ArgNameAndIndex("y", 0);
    const float height_scale = ctx->Attr<float>("scale_factor");
    const int64_t elem_cnt = y_tensor->shape().elem_cnt();
    const int64_t in_height = x_tensor->shape().At(2);
    const int64_t out_height = y_tensor->shape().At(2);
    if (in_height == out_height) {
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), y_tensor->mut_dptr<void>(), x_tensor->dptr<void>(),
          x_tensor->shape().elem_cnt() * GetSizeOfDataType(x_tensor->data_type()));
    } else {
      NdIndexOffsetHelper<int64_t, 3> in_helper(x_tensor->shape().At(0), x_tensor->shape().At(1),
                                                x_tensor->shape().At(2));
      NdIndexOffsetHelper<int64_t, 3> out_helper(y_tensor->shape().At(0), y_tensor->shape().At(1),
                                                 y_tensor->shape().At(2));
      RUN_CUDA_KERNEL((UpsampleNearest1DForward<T>), ctx->stream(), elem_cnt, elem_cnt,
                      x_tensor->dptr<T>(), in_helper, out_helper, x_tensor->shape().At(2),
                      1.f / height_scale, y_tensor->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleNearestGrad1DGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearestGrad1DGPUKernel() = default;
  ~UpsampleNearestGrad1DGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);

    Memset<DeviceType::kCUDA>(ctx->stream(), dx_tensor->mut_dptr<T>(), 0,
                              dx_tensor->shape().elem_cnt() * sizeof(T));
    const user_op::Tensor* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const float height_scale = ctx->Attr<float>("scale_factor");
    const int64_t elem_cnt = dy_tensor->shape().elem_cnt();
    const int64_t in_height = dx_tensor->shape().At(2);
    const int64_t out_height = dy_tensor->shape().At(2);
    if (in_height == out_height) {
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), dx_tensor->mut_dptr<void>(), dy_tensor->dptr<void>(),
          dy_tensor->shape().elem_cnt() * GetSizeOfDataType(dy_tensor->data_type()));
    } else {
      NdIndexOffsetHelper<int64_t, 3> dy_helper(dy_tensor->shape().At(0), dy_tensor->shape().At(1),
                                                dy_tensor->shape().At(2));
      NdIndexOffsetHelper<int64_t, 3> dx_helper(dx_tensor->shape().At(0), dx_tensor->shape().At(1),
                                                dx_tensor->shape().At(2));
      RUN_CUDA_KERNEL((UpsampleNearest1DBackward<T>), ctx->stream(), elem_cnt, elem_cnt,
                      dy_tensor->dptr<T>(), dy_helper, dx_helper, dx_tensor->shape().At(2),
                      1.f / height_scale, dx_tensor->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPNEAREST1D_CUDA_KERNEL(dtype)                                     \
  REGISTER_USER_KERNEL("upsample_nearest_1d")                                           \
      .SetCreateFn<UpsampleNearest1DGPUKernel<dtype>>()                                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)); \
  REGISTER_USER_KERNEL("upsample_nearest_1d_grad")                                      \
      .SetCreateFn<UpsampleNearestGrad1DGPUKernel<dtype>>()                             \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_UPSAMPNEAREST1D_CUDA_KERNEL(float)
REGISTER_UPSAMPNEAREST1D_CUDA_KERNEL(double)

template<typename T>
class UpsampleNearest2DGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearest2DGPUKernel() = default;
  ~UpsampleNearest2DGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_tensor = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_tensor = ctx->Tensor4ArgNameAndIndex("y", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const int64_t elem_cnt = y_tensor->shape().elem_cnt();

    const int64_t in_height = x_tensor->shape().At(2);
    const int64_t in_width = x_tensor->shape().At(3);
    const int64_t out_height = y_tensor->shape().At(2);
    const int64_t out_width = y_tensor->shape().At(3);
    if (in_height == out_height && in_width == out_width) {
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), y_tensor->mut_dptr<void>(), x_tensor->dptr<void>(),
          x_tensor->shape().elem_cnt() * GetSizeOfDataType(x_tensor->data_type()));
    } else {
      NdIndexOffsetHelper<int64_t, 4> in_helper(x_tensor->shape().At(0), x_tensor->shape().At(1),
                                                x_tensor->shape().At(2), x_tensor->shape().At(3));
      NdIndexOffsetHelper<int64_t, 4> out_helper(y_tensor->shape().At(0), y_tensor->shape().At(1),
                                                 y_tensor->shape().At(2), y_tensor->shape().At(3));
      RUN_CUDA_KERNEL((UpsampleNearest2DForward<T>), ctx->stream(), elem_cnt, elem_cnt,
                      x_tensor->dptr<T>(), in_helper, out_helper, x_tensor->shape().At(2),
                      x_tensor->shape().At(3), 1.f / height_scale, 1.f / width_scale,
                      y_tensor->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleNearest2DGradGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearest2DGradGPUKernel() = default;
  ~UpsampleNearest2DGradGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);

    Memset<DeviceType::kCUDA>(ctx->stream(), dx_tensor->mut_dptr<T>(), 0,
                              dx_tensor->shape().elem_cnt() * sizeof(T));
    const user_op::Tensor* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const int64_t elem_cnt = dy_tensor->shape().elem_cnt();
    const int64_t in_height = dx_tensor->shape().At(2);
    const int64_t in_width = dx_tensor->shape().At(3);
    const int64_t out_height = dy_tensor->shape().At(2);
    const int64_t out_width = dy_tensor->shape().At(3);
    if (in_height == out_height && in_width == out_width) {
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), dx_tensor->mut_dptr<void>(), dy_tensor->dptr<void>(),
          dy_tensor->shape().elem_cnt() * GetSizeOfDataType(dy_tensor->data_type()));
    } else {
      NdIndexOffsetHelper<int64_t, 4> dy_helper(dy_tensor->shape().At(0), dy_tensor->shape().At(1),
                                                dy_tensor->shape().At(2), dy_tensor->shape().At(3));
      NdIndexOffsetHelper<int64_t, 4> dx_helper(dx_tensor->shape().At(0), dx_tensor->shape().At(1),
                                                dx_tensor->shape().At(2), dx_tensor->shape().At(3));
      RUN_CUDA_KERNEL((UpsampleNearest2DBackward<T>), ctx->stream(), elem_cnt, elem_cnt,
                      dy_tensor->dptr<T>(), dy_helper, dx_helper, dx_tensor->shape().At(2),
                      dx_tensor->shape().At(3), 1.f / height_scale, 1.f / width_scale,
                      dx_tensor->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPLE_NEAREST_2D_CUDA_KERNEL(dtype)                                 \
  REGISTER_USER_KERNEL("upsample_nearest_2d")                                           \
      .SetCreateFn<UpsampleNearest2DGPUKernel<dtype>>()                                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)); \
  REGISTER_USER_KERNEL("upsample_nearest_2d_grad")                                      \
      .SetCreateFn<UpsampleNearest2DGradGPUKernel<dtype>>()                             \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_UPSAMPLE_NEAREST_2D_CUDA_KERNEL(float)
REGISTER_UPSAMPLE_NEAREST_2D_CUDA_KERNEL(double)

template<typename T>
class UpsampleNearest3DGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearest3DGPUKernel() = default;
  ~UpsampleNearest3DGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_tensor = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_tensor = ctx->Tensor4ArgNameAndIndex("y", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const float depth_scale = ctx->Attr<float>("depth_scale");
    const int64_t elem_cnt = y_tensor->shape().elem_cnt();
    NdIndexOffsetHelper<int64_t, 5> in_helper(x_tensor->shape().At(0), x_tensor->shape().At(1),
                                              x_tensor->shape().At(2), x_tensor->shape().At(3),
                                              x_tensor->shape().At(4));
    NdIndexOffsetHelper<int64_t, 5> out_helper(y_tensor->shape().At(0), y_tensor->shape().At(1),
                                               y_tensor->shape().At(2), y_tensor->shape().At(3),
                                               y_tensor->shape().At(4));
    RUN_CUDA_KERNEL((UpsampleNearest3DForward<T>), ctx->stream(), elem_cnt, elem_cnt,
                    x_tensor->dptr<T>(), in_helper, out_helper, x_tensor->shape().At(2),
                    x_tensor->shape().At(3), x_tensor->shape().At(4), 1.f / depth_scale,
                    1.f / height_scale, 1.f / width_scale, y_tensor->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleNearestGrad3DGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearestGrad3DGPUKernel() = default;
  ~UpsampleNearestGrad3DGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);

    Memset<DeviceType::kCUDA>(ctx->stream(), dx_tensor->mut_dptr<T>(), 0,
                              dx_tensor->shape().elem_cnt() * sizeof(T));
    const user_op::Tensor* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const float depth_scale = ctx->Attr<float>("depth_scale");
    const int64_t elem_cnt = dy_tensor->shape().elem_cnt();
    NdIndexOffsetHelper<int64_t, 5> dy_helper(dy_tensor->shape().At(0), dy_tensor->shape().At(1),
                                              dy_tensor->shape().At(2), dy_tensor->shape().At(3),
                                              dy_tensor->shape().At(4));
    NdIndexOffsetHelper<int64_t, 5> dx_helper(dx_tensor->shape().At(0), dx_tensor->shape().At(1),
                                              dx_tensor->shape().At(2), dx_tensor->shape().At(3),
                                              dx_tensor->shape().At(4));
    RUN_CUDA_KERNEL((UpsampleNearest3DBackward<T>), ctx->stream(), elem_cnt, elem_cnt,
                    dy_tensor->dptr<T>(), dy_helper, dx_helper, dx_tensor->shape().At(2),
                    dx_tensor->shape().At(3), dx_tensor->shape().At(4), 1.f / depth_scale,
                    1.f / height_scale, 1.f / width_scale, dx_tensor->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPNEAREST3D_CUDA_KERNEL(dtype)                                     \
  REGISTER_USER_KERNEL("upsample_nearest_3d")                                           \
      .SetCreateFn<UpsampleNearest3DGPUKernel<dtype>>()                                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)); \
  REGISTER_USER_KERNEL("upsample_nearest_3d_grad")                                      \
      .SetCreateFn<UpsampleNearestGrad3DGPUKernel<dtype>>()                             \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_UPSAMPNEAREST3D_CUDA_KERNEL(float)
REGISTER_UPSAMPNEAREST3D_CUDA_KERNEL(double)

}  // namespace oneflow
