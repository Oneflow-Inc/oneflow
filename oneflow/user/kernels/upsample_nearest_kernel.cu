#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/common/nd_index_offset_helper.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include "oneflow/user/kernels/upsample_kernel.h"
#include "oneflow/core/kernel/cuda_graph_support.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void UpsampleNearest1DForward(const int64_t elem_cnt, const T* in_dptr,
                                         NdIndexOffsetHelper<int64_t, 3> in_helper,
                                         NdIndexOffsetHelper<int64_t, 3> out_helper,
                                         const int64_t in_height, const double scale_factor,
                                         T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h;
    out_helper.OffsetToNdIndex(index, n, c, h);
    const int64_t in_h = GetNearestInputIndex(h, scale_factor, in_height);
    out_dptr[index] = in_dptr[in_helper.NdIndexToOffset(n, c, in_h)];
  }
}

template<typename T>
__global__ void UpsampleNearest1DBackward(const int64_t elem_cnt, const T* dy_dptr,
                                          NdIndexOffsetHelper<int64_t, 3> dy_helper,
                                          NdIndexOffsetHelper<int64_t, 3> dx_helper,
                                          const int64_t in_height, const double scale_factor,
                                          T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h;
    dy_helper.OffsetToNdIndex(index, n, c, h);
    const int64_t dx_h = GetNearestInputIndex(h, scale_factor, in_height);
    cuda::atomic::FastAdd(dx_dptr, dx_helper.NdIndexToOffset(n, c, dx_h), elem_cnt,
                          static_cast<T>(dy_dptr[index]));
  }
}

template<typename T>
__global__ void UpsampleNearest2DForward(const int64_t elem_cnt, const T* in_dptr,
                                         NdIndexOffsetHelper<int64_t, 4> in_helper,
                                         NdIndexOffsetHelper<int64_t, 4> out_helper,
                                         const int64_t in_height, const int64_t in_width,
                                         const double scale_h, const double scale_w, T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    out_helper.OffsetToNdIndex(index, n, c, h, w);
    const int64_t in_h = GetNearestInputIndex(h, scale_h, in_height);
    const int64_t in_w = GetNearestInputIndex(w, scale_w, in_width);
    out_dptr[index] = in_dptr[in_helper.NdIndexToOffset(n, c, in_h, in_w)];
  }
}

template<typename T>
struct alignas(2 * sizeof(T)) Pack2X {
  T x;
  T y;
};

template<typename T>
__global__ void UpsampleNearest2D2XForward(const int32_t in_elem_cnt, const T* in_dptr,
                                           const int32_t in_height, const int32_t in_width,
                                           T* out_dptr) {
  const int32_t in_hw_size = in_width * in_height;
  CUDA_1D_KERNEL_LOOP(index, in_elem_cnt) {
    const T in_value = in_dptr[index];
    const int32_t nc_idx = index / in_hw_size;
    const int32_t hw_off = index - nc_idx * in_hw_size;
    const int32_t h = hw_off / in_width;
    const int32_t w = hw_off - h * in_width;
    Pack2X<T> out_value{in_value, in_value};
    Pack2X<T>* out_pack_dptr = reinterpret_cast<Pack2X<T>*>(out_dptr);
    out_pack_dptr[nc_idx * in_hw_size * 2 + h * 2 * in_width + w] = out_value;
    out_pack_dptr[nc_idx * in_hw_size * 2 + (h * 2 + 1) * in_width + w] = out_value;
  }
}

template<typename T>
__global__ void UpsampleNearest2DBackward(const int64_t elem_cnt, const T* dy_dptr,
                                          NdIndexOffsetHelper<int64_t, 4> dy_helper,
                                          NdIndexOffsetHelper<int64_t, 4> dx_helper,
                                          const int64_t dx_height, const int64_t dx_width,
                                          const double scale_h, const double scale_w, T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    dy_helper.OffsetToNdIndex(index, n, c, h, w);
    const int64_t dx_h = GetNearestInputIndex(h, scale_h, dx_height);
    const int64_t dx_w = GetNearestInputIndex(w, scale_w, dx_width);
    cuda::atomic::FastAdd(dx_dptr, dx_helper.NdIndexToOffset(n, c, dx_h, dx_w), elem_cnt,
                          static_cast<T>(dy_dptr[index]));
  }
}

template<typename T>
__global__ void UpsampleNearest2D2XBackward(const int32_t in_elem_cnt, const T* dy_dptr,
                                            const int32_t dx_height, const int32_t dx_width,
                                            T* dx_dptr) {
  const int32_t dx_hw_size = dx_height * dx_width;
  CUDA_1D_KERNEL_LOOP(index, in_elem_cnt) {
    T dx_value = 0.0;
    const int32_t nc_idx = index / dx_hw_size;
    const int32_t dx_hw_off = index - nc_idx * dx_hw_size;
    const int32_t dx_h = dx_hw_off / dx_width;
    const int32_t dx_w = dx_hw_off - dx_h * dx_width;
    const Pack2X<T>* dy_pack_dptr = reinterpret_cast<const Pack2X<T>*>(dy_dptr);
    const Pack2X<T> dy_pack_value1 =
        dy_pack_dptr[nc_idx * dx_hw_size * 2 + dx_h * 2 * dx_width + dx_w];
    const Pack2X<T> dy_pack_value2 =
        dy_pack_dptr[nc_idx * dx_hw_size * 2 + (dx_h * 2 + 1) * dx_width + dx_w];
    dx_value += dy_pack_value1.x;
    dx_value += dy_pack_value1.y;
    dx_value += dy_pack_value2.x;
    dx_value += dy_pack_value2.y;
    dx_dptr[index] = dx_value;
  }
}

template<typename T>
__global__ void UpsampleNearest3DForward(const int64_t elem_cnt, const T* in_dptr,
                                         NdIndexOffsetHelper<int64_t, 5> in_helper,
                                         NdIndexOffsetHelper<int64_t, 5> out_helper,
                                         const int64_t in_depth, const int64_t in_height,
                                         const int64_t in_width, const float scale_d,
                                         const float scale_h, const float scale_w, T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, d, h, w;
    out_helper.OffsetToNdIndex(index, n, c, d, h, w);
    const int64_t in_h = GetNearestInputIndex(h, scale_h, in_height);
    const int64_t in_w = GetNearestInputIndex(w, scale_w, in_width);
    const int64_t in_d = GetNearestInputIndex(d, scale_d, in_depth);
    out_dptr[index] = in_dptr[in_helper.NdIndexToOffset(n, c, in_d, in_h, in_w)];
  }
}

template<typename T>
__global__ void UpsampleNearest3DBackward(const int64_t elem_cnt, const T* dy_dptr,
                                          NdIndexOffsetHelper<int64_t, 5> dy_helper,
                                          NdIndexOffsetHelper<int64_t, 5> dx_helper,
                                          const int64_t in_depth, const int64_t in_height,
                                          const int64_t in_width, const float scale_d,
                                          const float scale_h, const float scale_w, T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, d, h, w;
    dy_helper.OffsetToNdIndex(index, n, c, d, h, w);
    const int64_t dx_h = GetNearestInputIndex(h, scale_h, in_height);
    const int64_t dx_w = GetNearestInputIndex(w, scale_w, in_width);
    const int64_t in_d = GetNearestInputIndex(d, scale_d, in_depth);
    cuda::atomic::FastAdd(dx_dptr, dx_helper.NdIndexToOffset(n, c, in_d, dx_h, dx_w), elem_cnt,
                          static_cast<T>(dy_dptr[index]));
  }
}

}  // namespace

template<typename T>
class UpsampleNearest1DGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearest1DGPUKernel() = default;
  ~UpsampleNearest1DGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_tensor = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_tensor = ctx->Tensor4ArgNameAndIndex("y", 0);
    const std::vector<int64_t> output_size = ctx->Attr<std::vector<int64_t>>("output_size");
    double height_scale = ctx->Attr<double>("scale_factor");
    const int64_t elem_cnt = y_tensor->shape_view().elem_cnt();
    const int64_t in_height = x_tensor->shape_view().At(2);
    const int64_t out_height = y_tensor->shape_view().At(2);
    if (!output_size.empty()) {
      height_scale = static_cast<double>(out_height) / static_cast<double>(in_height);
    }
    if (in_height == out_height) {
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), y_tensor->mut_dptr<void>(), x_tensor->dptr<void>(),
          x_tensor->shape_view().elem_cnt() * GetSizeOfDataType(x_tensor->data_type()));
    } else {
      NdIndexOffsetHelper<int64_t, 3> in_helper(
          x_tensor->shape_view().At(0), x_tensor->shape_view().At(1), x_tensor->shape_view().At(2));
      NdIndexOffsetHelper<int64_t, 3> out_helper(
          y_tensor->shape_view().At(0), y_tensor->shape_view().At(1), y_tensor->shape_view().At(2));
      RUN_CUDA_KERNEL((UpsampleNearest1DForward<T>), ctx->stream(), elem_cnt, elem_cnt,
                      x_tensor->dptr<T>(), in_helper, out_helper, x_tensor->shape_view().At(2),
                      1.f / height_scale, y_tensor->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleNearestGrad1DGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearestGrad1DGPUKernel() = default;
  ~UpsampleNearestGrad1DGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);

    const user_op::Tensor* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const std::vector<int64_t> output_size = ctx->Attr<std::vector<int64_t>>("output_size");
    double height_scale = ctx->Attr<double>("scale_factor");
    const int64_t elem_cnt = dy_tensor->shape_view().elem_cnt();
    const int64_t in_height = dx_tensor->shape_view().At(2);
    const int64_t out_height = dy_tensor->shape_view().At(2);
    if (!output_size.empty()) {
      height_scale = static_cast<double>(out_height) / static_cast<double>(in_height);
    }
    if (in_height == out_height) {
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), dx_tensor->mut_dptr<void>(), dy_tensor->dptr<void>(),
          dy_tensor->shape_view().elem_cnt() * GetSizeOfDataType(dy_tensor->data_type()));
    } else {
      Memset<DeviceType::kCUDA>(ctx->stream(), dx_tensor->mut_dptr<T>(), 0,
                                dx_tensor->shape_view().elem_cnt() * sizeof(T));
      NdIndexOffsetHelper<int64_t, 3> dy_helper(dy_tensor->shape_view().At(0),
                                                dy_tensor->shape_view().At(1),
                                                dy_tensor->shape_view().At(2));
      NdIndexOffsetHelper<int64_t, 3> dx_helper(dx_tensor->shape_view().At(0),
                                                dx_tensor->shape_view().At(1),
                                                dx_tensor->shape_view().At(2));
      RUN_CUDA_KERNEL((UpsampleNearest1DBackward<T>), ctx->stream(), elem_cnt, elem_cnt,
                      dy_tensor->dptr<T>(), dy_helper, dx_helper, dx_tensor->shape_view().At(2),
                      1.f / height_scale, dx_tensor->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPNEAREST1D_CUDA_KERNEL(dtype)                                     \
  REGISTER_USER_KERNEL("upsample_nearest_1d")                                           \
      .SetCreateFn<UpsampleNearest1DGPUKernel<dtype>>()                                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)); \
  REGISTER_USER_KERNEL("upsample_nearest_1d_grad")                                      \
      .SetCreateFn<UpsampleNearestGrad1DGPUKernel<dtype>>()                             \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_UPSAMPNEAREST1D_CUDA_KERNEL(float)
REGISTER_UPSAMPNEAREST1D_CUDA_KERNEL(double)

template<typename T>
class UpsampleNearest2DGPUKernel final : public user_op::OpKernel,
                                         public user_op::CudaGraphSupport {
 public:
  UpsampleNearest2DGPUKernel() = default;
  ~UpsampleNearest2DGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_tensor = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_tensor = ctx->Tensor4ArgNameAndIndex("y", 0);
    const std::vector<int64_t> output_size = ctx->Attr<std::vector<int64_t>>("output_size");
    double height_scale = ctx->Attr<double>("height_scale");
    double width_scale = ctx->Attr<double>("width_scale");
    const int64_t out_elem_cnt = y_tensor->shape_view().elem_cnt();
    const int64_t in_elem_cnt = x_tensor->shape_view().elem_cnt();
    const int64_t in_height = x_tensor->shape_view().At(2);
    const int64_t in_width = x_tensor->shape_view().At(3);
    const int64_t out_height = y_tensor->shape_view().At(2);
    const int64_t out_width = y_tensor->shape_view().At(3);
    if (!output_size.empty()) {
      height_scale = static_cast<double>(out_height) / static_cast<double>(in_height);
      width_scale = static_cast<double>(out_width) / static_cast<double>(in_width);
    }

    if (in_height == out_height && in_width == out_width) {
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), y_tensor->mut_dptr<void>(), x_tensor->dptr<void>(),
          x_tensor->shape_view().elem_cnt() * GetSizeOfDataType(x_tensor->data_type()));
    } else {
      const int64_t n = x_tensor->shape_view().At(0);
      const int64_t c = x_tensor->shape_view().At(1);
      if (out_height == 2 * in_height && out_width == 2 * in_width && in_elem_cnt <= 1 << 29) {
        RUN_CUDA_KERNEL(UpsampleNearest2D2XForward<T>, ctx->stream(), in_elem_cnt, in_elem_cnt,
                        x_tensor->dptr<T>(), in_height, in_width, y_tensor->mut_dptr<T>());
      } else {
        NdIndexOffsetHelper<int64_t, 4> in_helper(n, c, in_height, in_width);
        NdIndexOffsetHelper<int64_t, 4> out_helper(n, c, out_height, out_width);
        RUN_CUDA_KERNEL((UpsampleNearest2DForward<T>), ctx->stream(), out_elem_cnt, out_elem_cnt,
                        x_tensor->dptr<T>(), in_helper, out_helper, in_height, in_width,
                        1.f / height_scale, 1.f / width_scale, y_tensor->mut_dptr<T>());
      }
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleNearest2DGradGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearest2DGradGPUKernel() = default;
  ~UpsampleNearest2DGradGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);

    const user_op::Tensor* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const std::vector<int64_t> output_size = ctx->Attr<std::vector<int64_t>>("output_size");
    double height_scale = ctx->Attr<double>("height_scale");
    double width_scale = ctx->Attr<double>("width_scale");
    const int64_t elem_cnt = dy_tensor->shape_view().elem_cnt();
    const int64_t in_elem_cnt = dx_tensor->shape_view().elem_cnt();
    const int64_t in_height = dx_tensor->shape_view().At(2);
    const int64_t in_width = dx_tensor->shape_view().At(3);
    const int64_t out_height = dy_tensor->shape_view().At(2);
    const int64_t out_width = dy_tensor->shape_view().At(3);
    if (!output_size.empty()) {
      height_scale = static_cast<double>(out_height) / static_cast<double>(in_height);
      width_scale = static_cast<double>(out_width) / static_cast<double>(in_width);
    }
    if (in_height == out_height && in_width == out_width) {
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), dx_tensor->mut_dptr<void>(), dy_tensor->dptr<void>(),
          dy_tensor->shape_view().elem_cnt() * GetSizeOfDataType(dy_tensor->data_type()));
    } else {
      if (out_height == 2 * in_height && out_width == 2 * in_width && in_elem_cnt <= 1 << 29) {
        RUN_CUDA_KERNEL(UpsampleNearest2D2XBackward<T>, ctx->stream(), in_elem_cnt, in_elem_cnt,
                        dy_tensor->dptr<T>(), dx_tensor->shape_view().At(2),
                        dx_tensor->shape_view().At(3), dx_tensor->mut_dptr<T>());
      } else {
        Memset<DeviceType::kCUDA>(ctx->stream(), dx_tensor->mut_dptr<T>(), 0,
                                  dx_tensor->shape_view().elem_cnt() * sizeof(T));
        NdIndexOffsetHelper<int64_t, 4> dy_helper(
            dy_tensor->shape_view().At(0), dy_tensor->shape_view().At(1),
            dy_tensor->shape_view().At(2), dy_tensor->shape_view().At(3));
        NdIndexOffsetHelper<int64_t, 4> dx_helper(
            dx_tensor->shape_view().At(0), dx_tensor->shape_view().At(1),
            dx_tensor->shape_view().At(2), dx_tensor->shape_view().At(3));
        RUN_CUDA_KERNEL((UpsampleNearest2DBackward<T>), ctx->stream(), elem_cnt, elem_cnt,
                        dy_tensor->dptr<T>(), dy_helper, dx_helper, dx_tensor->shape_view().At(2),
                        dx_tensor->shape_view().At(3), 1.f / height_scale, 1.f / width_scale,
                        dx_tensor->mut_dptr<T>());
      }
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPLE_NEAREST_2D_CUDA_KERNEL(dtype)                                 \
  REGISTER_USER_KERNEL("upsample_nearest_2d")                                           \
      .SetCreateFn<UpsampleNearest2DGPUKernel<dtype>>()                                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)); \
  REGISTER_USER_KERNEL("upsample_nearest_2d_grad")                                      \
      .SetCreateFn<UpsampleNearest2DGradGPUKernel<dtype>>()                             \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_UPSAMPLE_NEAREST_2D_CUDA_KERNEL(float)
REGISTER_UPSAMPLE_NEAREST_2D_CUDA_KERNEL(half)
REGISTER_UPSAMPLE_NEAREST_2D_CUDA_KERNEL(double)

template<typename T>
class UpsampleNearest3DGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearest3DGPUKernel() = default;
  ~UpsampleNearest3DGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_tensor = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_tensor = ctx->Tensor4ArgNameAndIndex("y", 0);
    const std::vector<int64_t> output_size = ctx->Attr<std::vector<int64_t>>("output_size");
    double depth_scale = ctx->Attr<double>("depth_scale");
    double height_scale = ctx->Attr<double>("height_scale");
    double width_scale = ctx->Attr<double>("width_scale");
    const int64_t in_depth = x_tensor->shape_view().At(2);
    const int64_t in_height = x_tensor->shape_view().At(3);
    const int64_t in_width = x_tensor->shape_view().At(4);
    const int64_t out_depth = y_tensor->shape_view().At(2);
    const int64_t out_height = y_tensor->shape_view().At(3);
    const int64_t out_width = y_tensor->shape_view().At(4);
    const int64_t elem_cnt = y_tensor->shape_view().elem_cnt();
    if (!output_size.empty()) {
      depth_scale = static_cast<double>(out_depth) / static_cast<double>(in_depth);
      height_scale = static_cast<double>(out_height) / static_cast<double>(in_height);
      width_scale = static_cast<double>(out_width) / static_cast<double>(in_width);
    }
    NdIndexOffsetHelper<int64_t, 5> in_helper(
        x_tensor->shape_view().At(0), x_tensor->shape_view().At(1), x_tensor->shape_view().At(2),
        x_tensor->shape_view().At(3), x_tensor->shape_view().At(4));
    NdIndexOffsetHelper<int64_t, 5> out_helper(
        y_tensor->shape_view().At(0), y_tensor->shape_view().At(1), y_tensor->shape_view().At(2),
        y_tensor->shape_view().At(3), y_tensor->shape_view().At(4));
    RUN_CUDA_KERNEL((UpsampleNearest3DForward<T>), ctx->stream(), elem_cnt, elem_cnt,
                    x_tensor->dptr<T>(), in_helper, out_helper, x_tensor->shape_view().At(2),
                    x_tensor->shape_view().At(3), x_tensor->shape_view().At(4), 1.f / depth_scale,
                    1.f / height_scale, 1.f / width_scale, y_tensor->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleNearestGrad3DGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearestGrad3DGPUKernel() = default;
  ~UpsampleNearestGrad3DGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);

    Memset<DeviceType::kCUDA>(ctx->stream(), dx_tensor->mut_dptr<T>(), 0,
                              dx_tensor->shape_view().elem_cnt() * sizeof(T));
    const user_op::Tensor* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const std::vector<int64_t> output_size = ctx->Attr<std::vector<int64_t>>("output_size");
    double depth_scale = ctx->Attr<double>("depth_scale");
    double height_scale = ctx->Attr<double>("height_scale");
    double width_scale = ctx->Attr<double>("width_scale");
    const int64_t in_depth = dx_tensor->shape_view().At(2);
    const int64_t in_height = dx_tensor->shape_view().At(3);
    const int64_t in_width = dx_tensor->shape_view().At(4);
    const int64_t out_depth = dy_tensor->shape_view().At(2);
    const int64_t out_height = dy_tensor->shape_view().At(3);
    const int64_t out_width = dy_tensor->shape_view().At(4);
    const int64_t elem_cnt = dy_tensor->shape_view().elem_cnt();
    if (!output_size.empty()) {
      depth_scale = static_cast<double>(out_depth) / static_cast<double>(in_depth);
      height_scale = static_cast<double>(out_height) / static_cast<double>(in_height);
      width_scale = static_cast<double>(out_width) / static_cast<double>(in_width);
    }
    NdIndexOffsetHelper<int64_t, 5> dy_helper(
        dy_tensor->shape_view().At(0), dy_tensor->shape_view().At(1), dy_tensor->shape_view().At(2),
        dy_tensor->shape_view().At(3), dy_tensor->shape_view().At(4));
    NdIndexOffsetHelper<int64_t, 5> dx_helper(
        dx_tensor->shape_view().At(0), dx_tensor->shape_view().At(1), dx_tensor->shape_view().At(2),
        dx_tensor->shape_view().At(3), dx_tensor->shape_view().At(4));
    RUN_CUDA_KERNEL((UpsampleNearest3DBackward<T>), ctx->stream(), elem_cnt, elem_cnt,
                    dy_tensor->dptr<T>(), dy_helper, dx_helper, dx_tensor->shape_view().At(2),
                    dx_tensor->shape_view().At(3), dx_tensor->shape_view().At(4), 1.f / depth_scale,
                    1.f / height_scale, 1.f / width_scale, dx_tensor->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPNEAREST3D_CUDA_KERNEL(dtype)                                     \
  REGISTER_USER_KERNEL("upsample_nearest_3d")                                           \
      .SetCreateFn<UpsampleNearest3DGPUKernel<dtype>>()                                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)); \
  REGISTER_USER_KERNEL("upsample_nearest_3d_grad")                                      \
      .SetCreateFn<UpsampleNearestGrad3DGPUKernel<dtype>>()                             \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_UPSAMPNEAREST3D_CUDA_KERNEL(float)
REGISTER_UPSAMPNEAREST3D_CUDA_KERNEL(double)

}  // namespace oneflow
