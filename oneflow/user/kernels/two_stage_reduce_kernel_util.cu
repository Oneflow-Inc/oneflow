#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/two_stage_reduce_kernel_util.h"

namespace oneflow {

namespace {

template<typename T, typename K>
__global__ void DivideGpu(const int64_t n, const T* x, const K* count, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = x[i] / count[i]; }
}

template<typename T, typename K>
__global__ void MaskGpu(const int64_t n, const T* x, const K* mask, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = static_cast<T>(mask[i]) * x[i]; }
}

template<typename T, typename K>
__global__ void ScaleGpu(const int64_t n, const T* x, const K* scale, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = x[i] * scale[i]; }
}

}  // namespace

template<typename T, typename K>
struct TwoStageReduceKernelUtil<DeviceType::kGPU, T, K> {
  static void Divide(DeviceCtx* ctx, const int64_t n, const T* x, const K* count, T* y) {
    DivideGpu<T, K><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, x, count, y);
  }

  static void Mask(DeviceCtx* ctx, const int64_t n, const T* x, const K* mask, T* y) {
    MaskGpu<T, K><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, x, mask, y);
  }

  static void Scale(DeviceCtx* ctx, const int64_t n, const T* x, const K* scale, T* y) {
    ScaleGpu<T, K><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, x, scale, y);
  }
};

#define INSTANTIATE_TWO_STAGE_REDUCE_KERNEL_UTIL_GPU(data_type_pair, index_type_pair)          \
  template struct TwoStageReduceKernelUtil<DeviceType::kGPU, OF_PP_PAIR_FIRST(data_type_pair), \
                                           OF_PP_PAIR_FIRST(index_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_TWO_STAGE_REDUCE_KERNEL_UTIL_GPU,
                                 FLOATING_DATA_TYPE_SEQ INDEX_DATA_TYPE_SEQ, INT_DATA_TYPE_SEQ);
#undef INSTANTIATE_TWO_STAGE_REDUCE_KERNEL_UTIL_GPU

}  // namespace oneflow
