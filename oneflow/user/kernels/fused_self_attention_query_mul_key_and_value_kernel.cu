/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/slice_util.h"
#include "oneflow/core/kernel/new_kernel_util.h"

namespace oneflow {

namespace {

inline hipblasOperation_t GetCublasOp(char op) {
  switch (op) {
    case 'n':
    case 'N': {
      return HIPBLAS_OP_N;
    }
    case 't':
    case 'T': {
      return HIPBLAS_OP_T;
    }
    case 'c':
    case 'C': {
      return HIPBLAS_OP_C;
    }
    default: {
      UNIMPLEMENTED();
    }
  }
  return HIPBLAS_OP_N;
}

template<typename T>
struct CudaDataTypeTrait;

template<>
struct CudaDataTypeTrait<float> {
  const static hipDataType value = HIP_R_32F;
};

template<>
struct CudaDataTypeTrait<half> {
  const static hipDataType value = HIP_R_16F;
};

template<typename T>
void CublasBatchGemm(hipblasHandle_t handle, char transa, char transb, int64_t m, int64_t n,
                     int64_t k, T alpha, const T* a, int64_t lda, int64_t stridea, const T* b,
                     int64_t ldb, int64_t strideb, T beta, T* c, int64_t ldc, int64_t stridec,
                     int64_t batch_size) {
  hipblasOperation_t opa = GetCublasOp(transa);
  hipblasOperation_t opb = GetCublasOp(transb);
  if (CUDA_VERSION >= 9010 && GetCudaSmVersion() >= 500) {
#if CUDA_VERSION >= 9010
    hipDataType data_type = CudaDataTypeTrait<T>::value;
    OF_CUBLAS_CHECK(hipblasGemmStridedBatchedEx(
        handle, opa, opb, m, n, k, reinterpret_cast<const void*>(&alpha),
        reinterpret_cast<const void*>(a), data_type, lda, stridea, reinterpret_cast<const void*>(b),
        data_type, ldb, strideb, reinterpret_cast<const void*>(&beta), reinterpret_cast<void*>(c),
        data_type, ldc, stridec, batch_size, data_type, HIPBLAS_GEMM_DEFAULT));
#else
    UNIMPLEMENTED();
#endif
  } else {
    cublas_gemmStridedBatched<T>(handle, opa, opb, m, n, k, &alpha, a, ldb, stridea, b, ldb,
                                 strideb, &beta, c, ldc, stridec, batch_size);
  }
}

#if CUDA_VERSION >= 9010

template<>
void CublasBatchGemm<half>(hipblasHandle_t handle, char transa, char transb, int64_t m, int64_t n,
                           int64_t k, half alpha, const half* a, int64_t lda, int64_t stridea,
                           const half* b, int64_t ldb, int64_t strideb, half beta, half* c,
                           int64_t ldc, int64_t stridec, int64_t batch_size) {
  using comp_t = float;
  hipblasOperation_t opa = GetCublasOp(transa);
  hipblasOperation_t opb = GetCublasOp(transb);

  if (GetCudaSmVersion() >= 500) {
    float alpha_f = static_cast<comp_t>(alpha);
    float beta_f = static_cast<comp_t>(beta);
#if CUDA_VERSION >= 11000
    hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;
#else
    hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
#endif
    hipDataType data_type = CudaDataTypeTrait<half>::value;
    hipDataType comp_type = CudaDataTypeTrait<comp_t>::value;
    OF_CUBLAS_CHECK(hipblasGemmStridedBatchedEx(
        handle, opa, opb, m, n, k, &alpha_f, reinterpret_cast<const void*>(a), data_type, lda,
        stridea, reinterpret_cast<const void*>(b), data_type, ldb, strideb, &beta_f,
        reinterpret_cast<void*>(c), data_type, ldc, stridec, batch_size, comp_type, algo));
  } else {
    cublas_gemmStridedBatched<half>(handle, opa, opb, m, n, k, &alpha, a, lda, stridea, b, ldb,
                                    strideb, &beta, c, ldc, stridec, batch_size);
  }
}

template<>
void CublasBatchGemm<float16>(hipblasHandle_t handle, char transa, char transb, int64_t m, int64_t n,
                              int64_t k, float16 alpha, const float16* a, int64_t lda,
                              int64_t stridea, const float16* b, int64_t ldb, int64_t strideb,
                              float16 beta, float16* c, int64_t ldc, int64_t stridec,
                              int64_t batch_size) {
  CublasBatchGemm<half>(handle, transa, transb, m, n, k, static_cast<half>(alpha),
                        reinterpret_cast<const half*>(a), lda, stridea,
                        reinterpret_cast<const half*>(b), ldb, strideb, static_cast<half>(beta),
                        reinterpret_cast<half*>(c), ldc, stridec, batch_size);
}

#endif  // CUDA_VERSION >= 9010

template<typename T>
void BatchedGemm(DeviceCtx* ctx, char opa, char opb, int64_t m, int64_t n, int64_t k, float alpha,
                 const T* a, int64_t lda, int64_t stridea, const T* b, int64_t ldb, int64_t strideb,
                 float beta, T* c, int64_t ldc, int64_t stridec, int64_t batch_size) {
  // swap m and n, a and b to convert from row-major to col-major
  CublasBatchGemm<T>(ctx->cublas_pmh_handle(), opb, opa, n, m, k, static_cast<T>(alpha), b, ldb,
                     strideb, a, lda, stridea, static_cast<T>(beta), c, ldc, stridec, batch_size);
}

SliceParams ConstructSliceParams4Value(int64_t seq_len, int64_t batch_size, int64_t num_heads,
                                       int64_t head_size) {
  // slice (s, b, n, 3, h) to (s, b, n, 1, h)
  SliceParams params;
  std::memset(&params, 0, sizeof(SliceParams));
  params.ndim = 4;
  params.dims[0] = seq_len;
  params.dims[1] = batch_size;
  params.dims[2] = num_heads;
  params.dims[3] = 3 * head_size;
  params.start[0] = 0;
  params.start[1] = 0;
  params.start[2] = 0;
  params.start[3] = 2 * head_size;
  params.step[0] = 1;
  params.step[1] = 1;
  params.step[2] = 1;
  params.step[3] = 1;
  params.size[0] = seq_len;
  params.size[1] = batch_size;
  params.size[2] = num_heads;
  params.size[3] = head_size;
  return params;
}

template<typename T>
void TransposeGpu(DeviceCtx* ctx, const ShapeView& in_shape, const ShapeView& out_shape,
                  const std::vector<int32_t>& perm, const T* in, T* out) {
  CHECK_EQ(in_shape.NumAxes(), out_shape.NumAxes());
  int32_t num_axes = in_shape.NumAxes();
  CHECK_EQ(num_axes, perm.size());
  for (int i = 0; i < perm.size(); ++i) { CHECK_EQ(in_shape.At(perm[i]), out_shape.At(i)); }
  int64_t elem_cnt = in_shape.elem_cnt();
  NewKernelUtil<DeviceType::kGPU>::Transpose(ctx, num_axes, in_shape, out_shape, perm, elem_cnt, in,
                                             out);
}

template<typename T>
class FusedSelfAttentionQueryMulKeyAndValueGpuKernel final : public user_op::OpKernel {
 public:
  FusedSelfAttentionQueryMulKeyAndValueGpuKernel() = default;
  ~FusedSelfAttentionQueryMulKeyAndValueGpuKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* h_tensor = ctx->Tensor4ArgNameAndIndex("hidden_states", 0);
    int64_t seq_len = h_tensor->shape().At(0);
    int64_t batch_size = h_tensor->shape().At(1);
    int64_t hidden_size = h_tensor->shape().At(2);
    int64_t head_size = ctx->Attr<int64_t>("head_size");
    int64_t num_heads = hidden_size / (3 * head_size);
    int64_t ld = batch_size * hidden_size;
    int64_t stride = 3 * head_size;
    int64_t k_offset = head_size;

    // q * k: (sq, b, n, h) x (sk, b, n, h) => (b, n, sq, h) x (b, n, sk, h)
    // => (b, n, sq, h) x (b, n, h, sk) -> (b, n, sq, sk)
    float alpha = ctx->Attr<float>("alpha");
    user_op::Tensor* qmk_tensor = ctx->Tensor4ArgNameAndIndex("query_mul_key", 0);
    const T* q_dptr = h_tensor->dptr<T>();
    const T* k_dptr = h_tensor->dptr<T>() + k_offset;
    BatchedGemm<T>(ctx->device_ctx(), 'N', 'T', seq_len, seq_len, head_size, alpha, q_dptr, ld,
                   stride, k_dptr, ld, stride, 0.0f, qmk_tensor->mut_dptr<T>(), seq_len,
                   seq_len * seq_len, batch_size * num_heads);

    // slice v
    user_op::Tensor* tmp_v_tensor = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    user_op::Tensor* v_tensor = ctx->Tensor4ArgNameAndIndex("value", 0);
    SliceParams params = ConstructSliceParams4Value(seq_len, batch_size, num_heads, head_size);
    SliceKernelUtil<DeviceType::kGPU, T>::Forward(ctx->device_ctx(), params, h_tensor->dptr<T>(),
                                                  tmp_v_tensor->mut_dptr<T>());
    // v from (s, b, n, h) transpose to (b, n, s, h)
    Shape value_shape({seq_len, batch_size, num_heads, head_size});
    TransposeGpu<T>(ctx->device_ctx(), value_shape, v_tensor->shape(), {1, 2, 0, 3},
                    tmp_v_tensor->dptr<T>(), v_tensor->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class FusedSelfAttentionQueryMulKeyAndValueGradGpuKernel final : public user_op::OpKernel {
 public:
  FusedSelfAttentionQueryMulKeyAndValueGradGpuKernel() = default;
  ~FusedSelfAttentionQueryMulKeyAndValueGradGpuKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* v_grad_tensor = ctx->Tensor4ArgNameAndIndex("value_grad", 0);
    const user_op::Tensor* qmk_grad_tensor = ctx->Tensor4ArgNameAndIndex("query_mul_key_grad", 0);
    const user_op::Tensor* h_tensor = ctx->Tensor4ArgNameAndIndex("hidden_states", 0);
    user_op::Tensor* tmp_v_tensor = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    user_op::Tensor* h_grad_tensor = ctx->Tensor4ArgNameAndIndex("hidden_states_grad", 0);

    float alpha = ctx->Attr<float>("alpha");
    int64_t seq_len = h_grad_tensor->shape().At(0);
    int64_t batch_size = h_grad_tensor->shape().At(1);
    int64_t hidden_size = h_grad_tensor->shape().At(2);
    int64_t num_heads = v_grad_tensor->shape().At(1);
    int64_t head_size = v_grad_tensor->shape().At(3);
    int64_t ld = batch_size * hidden_size;
    int64_t stride = 3 * head_size;
    CHECK_EQ(hidden_size, num_heads * stride);

    // transpose from (b, n, s, h) to (s, b, n, h)
    Shape value_shape({seq_len, batch_size, num_heads, head_size});
    TransposeGpu<T>(ctx->device_ctx(), v_grad_tensor->shape(), value_shape, {2, 0, 1, 3},
                    v_grad_tensor->dptr<T>(), tmp_v_tensor->mut_dptr<T>());
    // slice v grad
    SliceParams params = ConstructSliceParams4Value(seq_len, batch_size, num_heads, head_size);
    SliceKernelUtil<DeviceType::kGPU, T>::Backward(
        ctx->device_ctx(), params, tmp_v_tensor->dptr<T>(), h_grad_tensor->mut_dptr<T>());

    // grad_q = grad_qmk * k
    // (b, n, sq, sk) x (b, n, sk, h) -> (b, n, s, h) <= (s, b, n, h) <= (s, b, n, 3, h)
    const T* qmk_grad_dptr = qmk_grad_tensor->dptr<T>();
    const T* k_dptr = h_tensor->dptr<T>() + head_size;
    T* grad_q_dptr = h_grad_tensor->mut_dptr<T>();
    BatchedGemm<T>(ctx->device_ctx(), 'N', 'N', seq_len, head_size, seq_len, alpha, qmk_grad_dptr,
                   seq_len, seq_len * seq_len, k_dptr, ld, stride, 0.0f, grad_q_dptr, ld, stride,
                   batch_size * num_heads);
    // grad_k = grad_qmk * q
    // (b, n, sk, sq) x (b, n, sq, h) -> (b, n, sk, h) <= (s, b, n, h) <= (s, b, n, 3, h)
    const T* q_dptr = h_tensor->dptr<T>();
    T* grad_k_dptr = h_grad_tensor->mut_dptr<T>() + head_size;
    BatchedGemm<T>(ctx->device_ctx(), 'T', 'N', seq_len, head_size, seq_len, alpha, qmk_grad_dptr,
                   seq_len, seq_len * seq_len, q_dptr, ld, stride, 0.0f, grad_k_dptr, ld, stride,
                   batch_size * num_heads);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

size_t InferTmpBufferSize(user_op::InferContext* ctx) {
  const Shape* value_shape = ctx->OutputShape("value", 0);
  DataType value_dtype = *ctx->OutputDType("value", 0);
  return value_shape->elem_cnt() * GetSizeOfDataType(value_dtype);
}

size_t InferGradTmpBufferSize(user_op::InferContext* ctx) {
  const Shape& value_shape = ctx->InputShape("value_grad", 0);
  const DataType& value_dtype = ctx->InputDType("value_grad", 0);
  return value_shape.elem_cnt() * GetSizeOfDataType(value_dtype);
}

}  // namespace

#define REGISTER_FUSED_SELF_ATTENTION_QUERY_MUL_KEY_AND_VALUE_GPU_KERNEL(dtype)                   \
  REGISTER_USER_KERNEL("fused_self_attention_query_mul_key_and_value")                            \
      .SetCreateFn<FusedSelfAttentionQueryMulKeyAndValueGpuKernel<dtype>>()                       \
      .SetIsMatchedHob((user_op::HobDeviceTag() == DeviceType::kGPU)                              \
                       & (user_op::HobDataType("hidden_states", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn(InferTmpBufferSize);

#define REGISTER_FUSED_SELF_ATTENTION_QUERY_MUL_KEY_AND_VALUE_GRAD_GPU_KERNEL(dtype)              \
  REGISTER_USER_KERNEL("fused_self_attention_query_mul_key_and_value_grad")                       \
      .SetCreateFn<FusedSelfAttentionQueryMulKeyAndValueGradGpuKernel<dtype>>()                   \
      .SetIsMatchedHob((user_op::HobDeviceTag() == DeviceType::kGPU)                              \
                       & (user_op::HobDataType("hidden_states", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn(InferGradTmpBufferSize);

REGISTER_FUSED_SELF_ATTENTION_QUERY_MUL_KEY_AND_VALUE_GPU_KERNEL(float)
REGISTER_FUSED_SELF_ATTENTION_QUERY_MUL_KEY_AND_VALUE_GPU_KERNEL(float16)
REGISTER_FUSED_SELF_ATTENTION_QUERY_MUL_KEY_AND_VALUE_GRAD_GPU_KERNEL(float)
REGISTER_FUSED_SELF_ATTENTION_QUERY_MUL_KEY_AND_VALUE_GRAD_GPU_KERNEL(float16)

}  // namespace oneflow
