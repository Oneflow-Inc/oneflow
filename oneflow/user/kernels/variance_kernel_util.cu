#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <glog/logging.h>
#include "oneflow/user/kernels/variance_kernel_util.h"
#include "oneflow/core/cuda/layer_norm.cuh"

namespace oneflow {
namespace user_op {

namespace {
template<typename T>
__inline__ __device__ T Nan();

template<>
__inline__ __device__ float Nan<float>() {
  return HIP_NAN_F;
}

template<>
__inline__ __device__ double Nan<double>() {
  return HIP_NAN;
}

template<>
__inline__ __device__ half Nan<half>() {
  return half(HIP_NAN_F);
}

#if CUDA_VERSION >= 11000
template<>
__inline__ __device__ hip_bfloat16 Nan<hip_bfloat16>() {
  return hip_bfloat16(HIP_NAN_F);
}
#endif
}  // namespace

template<typename T, typename ComputeType>
__global__ void ComputeVarUsingWelfordWrapper(const T* in_ptr, T* out_ptr, const VarParam var_param,
                                              bool is_nan) {
  if (is_nan) {
    CUDA_1D_KERNEL_LOOP(i, var_param.parallel_num) { out_ptr[i] = Nan<T>(); }
  } else {
    CUDA_1D_KERNEL_LOOP(i, var_param.parallel_num) {
      const size_t input_offset = LinearIndex2Offset(
          i, var_param.dim_size_in_caxis, var_param.stride_in_caxis, var_param.caxis_size);
      ComputeVarUsingWelford<T, ComputeType>(&in_ptr[input_offset], &out_ptr[i], var_param);
    }
  }
}

namespace {
template<typename T, typename ComputeType>
inline __device__ void WelfordReduce(const T* in_ptr, ComputeType* mean, ComputeType* m2,
                                     ComputeType* count, const size_t total_elem_cnt,
                                     const size_t start, const size_t step) {
  ComputeType old_mean = 0.0;
  for (size_t i = start; i < total_elem_cnt; i += step) {
    ++(*count);
    old_mean = *mean;
    *mean += (static_cast<ComputeType>(in_ptr[i]) - *mean) / *count;
    *m2 += (static_cast<ComputeType>(in_ptr[i]) - *mean)
           * (static_cast<ComputeType>(in_ptr[i]) - old_mean);
  }
}

template<typename T>
inline __device__ void WelfordCombine(const T* b_mean, const T* b_m2, const T* b_count, T* mean,
                                      T* m2, T* count, const size_t total_elem_cnt,
                                      const size_t start, const size_t step) {
  for (size_t i = start; i < total_elem_cnt; i += step) {
    cuda::layer_norm::WelfordCombine(b_mean[i], b_m2[i], b_count[i], mean, m2, count);
  }
}
__device__ int32_t done_block_count = 0;
}  // namespace

template<typename T, typename ComputeType>
__global__ void ComputeVarScalarOut(const T* in_ptr, T* out_ptr, ComputeType* tmp_buffer_ptr,
                                    const VarParam var_param, bool is_nan) {
  if (is_nan) {
    if (blockIdx.x == 0 && threadIdx.x == 0) { *out_ptr = Nan<T>(); }
    return;
  }
  const size_t elems_per_block = var_param.elem_cnt / gridDim.x;
  const size_t elems_per_thread = elems_per_block / blockDim.x;
  // tail element number in block
  size_t tail_elems = elems_per_block % blockDim.x;

  ComputeType thread_mean = 0.0;
  ComputeType thread_m2 = 0.0;
  ComputeType thread_count = 0.0;
  // every thread deal it's elems
  if (elems_per_thread > 0) {
    const size_t block_offset = blockIdx.x * elems_per_block;
    WelfordReduce<T, ComputeType>(&in_ptr[block_offset], &thread_mean, &thread_m2, &thread_count,
                                  elems_per_block - tail_elems, threadIdx.x, blockDim.x);
  }
  // thread 0 of last block handles tail element between blocks
  if (blockIdx.x == gridDim.x - 1 && threadIdx.x == 0) {
    tail_elems += var_param.elem_cnt % gridDim.x;
  }
  // thread 0 deal tail elems
  if (tail_elems != 0 && threadIdx.x == 0) {
    const size_t tail_offset = blockIdx.x * elems_per_block + blockDim.x * elems_per_thread;
    WelfordReduce<T, ComputeType>(&in_ptr[tail_offset], &thread_mean, &thread_m2, &thread_count,
                                  tail_elems,
                                  /*tail start=*/0, /*step=*/1);
  }

  ComputeType block_mean = 0;
  ComputeType block_m2 = 0;
  ComputeType block_count = 0;
  cuda::layer_norm::WelfordBlockAllReduce<ComputeType>(thread_mean, thread_m2, thread_count,
                                                       &block_mean, &block_m2, &block_count);

  if (gridDim.x == 1) {
    if (threadIdx.x == 0) {
      *out_ptr =
          cuda::layer_norm::Div(block_m2, (var_param.unbiased ? block_count - 1 : block_count));
    }
    return;
  }

  ComputeType* tmp_mean_ptr = tmp_buffer_ptr;
  ComputeType* tmp_m2_ptr = &tmp_mean_ptr[gridDim.x];
  ComputeType* tmp_count_ptr = &tmp_m2_ptr[gridDim.x];
  if (threadIdx.x == 0) {
    tmp_mean_ptr[blockIdx.x] = block_mean;
    tmp_m2_ptr[blockIdx.x] = block_m2;
    tmp_count_ptr[blockIdx.x] = block_count;
  }
  __shared__ bool is_last_block;
  if (threadIdx.x == 0) { is_last_block = atomicAdd(&done_block_count, 1) == gridDim.x - 1; }
  __syncthreads();
  if (is_last_block) {
    ComputeType last_block_thread_mean = 0;
    ComputeType last_block_thread_m2 = 0;
    ComputeType last_block_thread_count = 0;
    const size_t welforddatas_per_thread = gridDim.x / blockDim.x;
    const size_t tail_welforddatas = gridDim.x % blockDim.x;

    if (welforddatas_per_thread > 0) {
      WelfordCombine(tmp_mean_ptr, tmp_m2_ptr, tmp_count_ptr, &last_block_thread_mean,
                     &last_block_thread_m2, &last_block_thread_count, gridDim.x - tail_welforddatas,
                     threadIdx.x, blockDim.x);
    }
    // thread 0 deal tail welford data
    if (tail_welforddatas != 0 && threadIdx.x == 0) {
      const size_t last_block_tail_offset = blockDim.x * welforddatas_per_thread;
      WelfordCombine(&tmp_mean_ptr[last_block_tail_offset], &tmp_m2_ptr[last_block_tail_offset],
                     &tmp_count_ptr[last_block_tail_offset], &last_block_thread_mean,
                     &last_block_thread_m2, &last_block_thread_count, tail_welforddatas,
                     /*tail start=*/0, /*step=*/1);
    }
    ComputeType final_mean = 0;
    ComputeType final_m2 = 0;
    ComputeType final_count = 0;
    cuda::layer_norm::WelfordBlockAllReduce<ComputeType>(
        last_block_thread_mean, last_block_thread_m2, last_block_thread_count, &final_mean,
        &final_m2, &final_count);
    if (threadIdx.x == 0) {
      *out_ptr =
          cuda::layer_norm::Div(final_m2, (var_param.unbiased ? final_count - 1 : final_count));
      done_block_count = 0;
    }
  }
}

template<typename T, typename ComputeType>
struct VarFunctor<DeviceType::kCUDA, T, ComputeType> final {
  void operator()(ep::Stream* stream, const T* in_ptr, T* out_ptr, ComputeType* tmp_buffer_ptr,
                  const VarParam var_param) {
    int grid_dim = 0;
    int block_dim = 0;
    SetGridDimAndBlockDim(var_param.elem_cnt, &grid_dim, &block_dim);
    if (var_param.parallel_num == 1) {
      ComputeVarScalarOut<T, ComputeType>
          <<<grid_dim, block_dim, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
              in_ptr, out_ptr, tmp_buffer_ptr, var_param, IsNanOut(var_param));
    } else {
      // if var_param.parallel_num is 0, do nothing, return 0-size tensor
      if (var_param.parallel_num == 0) { return; }
      RUN_CUDA_KERNEL((ComputeVarUsingWelfordWrapper<T, ComputeType>), stream,
                      var_param.parallel_num, in_ptr, out_ptr, var_param, IsNanOut(var_param));
    }
  }
};

template struct VarFunctor<DeviceType::kCUDA, float, double>;
template struct VarFunctor<DeviceType::kCUDA, double, double>;
template struct VarFunctor<DeviceType::kCUDA, half, double>;

#if CUDA_VERSION >= 11000
template struct VarFunctor<DeviceType::kCUDA, hip_bfloat16, double>;
#endif
}  // namespace user_op
}  // namespace oneflow
