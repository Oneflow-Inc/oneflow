#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/core/device/cuda_pseudo_bfloat16.h"
#include "oneflow/core/ep/include/device.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/user/kernels/op_kernel_wrapper.h"
#include "oneflow/user/kernels/dropout_kernel.h"
#include "oneflow/user/kernels/random_seed_util.h"

namespace oneflow {

namespace {

constexpr int32_t kVecSize = 4;
constexpr int32_t kBlockSize = 256;

template<typename T>
constexpr int32_t GetDropoutPackSize() {
  // For float, bfloat16, half.
  return 4;
};

template<>
constexpr int32_t GetDropoutPackSize<half2>() {
  return 2;
};

template<>
constexpr int32_t GetDropoutPackSize<double>() {
  return 2;
};

union RandPack4 {
  float4 storage;
  float elem[4];
};

template<typename T>
struct GetPack2Type {
  using T2 = typename std::aligned_storage<2 * sizeof(T), 2 * sizeof(T)>::type;
};

template<>
struct GetPack2Type<half> {
  using T2 = half2;
};

#if CUDA_VERSION >= 11000
template<>
struct GetPack2Type<hip_bfloat16> {
  using T2 = nv_bfloat162;
};
#endif

template<typename T>
using Pack2Type = typename GetPack2Type<T>::T2;

using H2PackType = typename std::aligned_storage<4 * sizeof(half), 4 * sizeof(half)>::type;

template<typename T>
union H2Pack {
  cuda::elementwise::Pack<T, 4> pack_storage;
  Pack2Type<T> h2[2];
  __device__ H2Pack() {
    // do nothing
  }
};

template<>
union H2Pack<half> {
  cuda::elementwise::Pack<half, 4> pack_storage;
  half2 h2[2];
  __device__ H2Pack() {
    // do nothing
  }
};

#if CUDA_VERSION >= 11000
template<>
union H2Pack<hip_bfloat16> {
  cuda::elementwise::Pack<hip_bfloat16, 4> pack_storage;
  nv_bfloat162 h2[2];
  __device__ H2Pack() {
    // do nothing
  }
};
#endif

template<typename T>
__device__ Pack2Type<T> Make2(float v);

template<>
__device__ Pack2Type<half> Make2<half>(float v) {
  return __float2half2_rn(v);
}

#if CUDA_VERSION >= 11000
template<>
__device__ Pack2Type<hip_bfloat16> Make2<hip_bfloat16>(float v) {
  return __float2bfloat162_rn(v);
}
#endif

#if CUDA_VERSION >= 11000
#define RETURN_VOID_IF_HALF                                                                        \
  typename std::enable_if_t<(std::is_same<T, half>::value || std::is_same<T, hip_bfloat16>::value), \
                            void>
#else
#define RETURN_VOID_IF_HALF typename std::enable_if_t<std::is_same<T, half>::value, void>
#endif
#define RETURN_VOID_IF_FLOAT typename std::enable_if_t<std::is_same<T, float>::value, void>
#define RETURN_VOID_IF_DOUBLE typename std::enable_if_t<std::is_same<T, double>::value, void>

template<typename T, int pack_size, bool tail, bool has_addend>
__global__ RETURN_VOID_IF_FLOAT FusedDropoutAddGpu(uint64_t seed, uint64_t offset,
                                                   const int64_t elem_cnt, float rate, float scale,
                                                   int64_t n_tail, const T* x, bool* mask,
                                                   const T* addend, T* y, const T* tail_x,
                                                   bool* tail_mask, const T* tail_addend,
                                                   T* tail_y) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, offset, &state);
  using LoadType = cuda::elementwise::PackType<T, pack_size>;
  using LoadPack = cuda::elementwise::Pack<T, pack_size>;
  using MaskType = cuda::elementwise::PackType<bool, pack_size>;
  using MaskPack = cuda::elementwise::Pack<bool, pack_size>;

  T t_scale = static_cast<T>(scale);
  RandPack4 rand_uniform_pack4;
  for (int64_t linear_index = global_thread_id * pack_size; linear_index < elem_cnt;
       linear_index += gridDim.x * blockDim.x * pack_size) {
    rand_uniform_pack4.storage = hiprand_uniform4(&state);

    const LoadType* x_load = reinterpret_cast<const LoadType*>(x + linear_index);
    LoadPack x_vec;
    x_vec.storage = *x_load;

    LoadPack addend_vec;
    if (has_addend) {
      const LoadType* addend_load = reinterpret_cast<const LoadType*>(addend + linear_index);
      addend_vec.storage = *addend_load;
    }

    MaskPack mask_vec;
    LoadPack y_vec;
#pragma unroll
    for (int i = 0; i < pack_size; i++) {
      mask_vec.elem[i] = rand_uniform_pack4.elem[i] > rate;
      T tmp_float_mask = static_cast<float>(mask_vec.elem[i]);
      y_vec.elem[i] = x_vec.elem[i] * tmp_float_mask * t_scale;
      if (has_addend) { y_vec.elem[i] += addend_vec.elem[i]; }
    }

    *(reinterpret_cast<LoadType*>(y + linear_index)) = y_vec.storage;
    *(reinterpret_cast<MaskType*>(mask + linear_index)) = mask_vec.storage;
  }

  if (tail && global_thread_id < n_tail) {
    const float rand_uniform = hiprand_uniform(&state);
    const bool mask_val = rand_uniform > rate;
    tail_mask[global_thread_id] = mask_val;
    T tmp_float_mask = static_cast<float>(mask_val);
    T tmp_tail_out = tail_x[global_thread_id] * tmp_float_mask * t_scale;
    if (has_addend) { tmp_tail_out += tail_addend[global_thread_id]; }
    tail_y[global_thread_id] = tmp_tail_out;
  }
}

template<typename T, int pack_size, bool tail, bool has_addend>
__global__ RETURN_VOID_IF_HALF FusedDropoutAddGpu(uint64_t seed, uint64_t offset,
                                                  const int64_t elem_cnt, float rate, float scale,
                                                  int64_t n_tail, const T* x, bool* mask,
                                                  const T* addend, T* y, const T* tail_x,
                                                  bool* tail_mask, const T* tail_addend,
                                                  T* tail_y) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, offset, &state);
  using LoadType = cuda::elementwise::PackType<T, pack_size>;
  using LoadPack = cuda::elementwise::Pack<T, pack_size>;
  using StoreType = cuda::elementwise::PackType<Pack2Type<T>, pack_size / 2>;
  using StorePack = cuda::elementwise::Pack<Pack2Type<T>, pack_size / 2>;
  using MaskType = cuda::elementwise::PackType<bool, pack_size>;
  using MaskPack = cuda::elementwise::Pack<bool, pack_size>;

  RandPack4 rand_uniform_pack4;
  Pack2Type<T> h2_scale = Make2<T>(scale);

  for (int64_t linear_index = global_thread_id * pack_size; linear_index < elem_cnt;
       linear_index += gridDim.x * blockDim.x * pack_size) {
    rand_uniform_pack4.storage = hiprand_uniform4(&state);
    const LoadType* x_load = reinterpret_cast<const LoadType*>(x + linear_index);
    H2Pack<T> x_vec{};
    x_vec.pack_storage.storage = *x_load;

    H2Pack<T> addend_vec{};
    if (has_addend) {
      const LoadType* addend_load = reinterpret_cast<const LoadType*>(addend + linear_index);
      addend_vec.pack_storage.storage = *addend_load;
    }

    MaskPack mask_vec;
    StorePack y_vec;
    StorePack one_or_zero_h2;

    mask_vec.elem[0] = rand_uniform_pack4.elem[0] > rate;
    float tmp_float_mask = static_cast<float>(mask_vec.elem[0]);
    one_or_zero_h2.elem[0].x = tmp_float_mask;
    mask_vec.elem[1] = rand_uniform_pack4.elem[1] > rate;
    tmp_float_mask = static_cast<float>(mask_vec.elem[1]);
    one_or_zero_h2.elem[0].y = tmp_float_mask;
    y_vec.elem[0] = __hmul2(__hmul2(x_vec.h2[0], one_or_zero_h2.elem[0]), h2_scale);

    mask_vec.elem[2] = rand_uniform_pack4.elem[2] > rate;
    tmp_float_mask = static_cast<float>(mask_vec.elem[2]);
    one_or_zero_h2.elem[1].x = tmp_float_mask;
    mask_vec.elem[3] = rand_uniform_pack4.elem[3] > rate;
    tmp_float_mask = static_cast<float>(mask_vec.elem[3]);
    one_or_zero_h2.elem[1].y = tmp_float_mask;
    y_vec.elem[1] = __hmul2(__hmul2(x_vec.h2[1], one_or_zero_h2.elem[1]), h2_scale);

    if (has_addend) {
      y_vec.elem[0] = __hadd2(y_vec.elem[0], addend_vec.h2[0]);
      y_vec.elem[1] = __hadd2(y_vec.elem[1], addend_vec.h2[1]);
    }

    *(reinterpret_cast<StoreType*>(y + linear_index)) = y_vec.storage;
    *(reinterpret_cast<MaskType*>(mask + linear_index)) = mask_vec.storage;
  }

  if (tail && global_thread_id < n_tail) {
    const float rand_uniform = hiprand_uniform(&state);
    const bool mask_val = rand_uniform > rate;
    tail_mask[global_thread_id] = mask_val;
    float tmp_half_mask = static_cast<float>(mask_val);
    T tmp_tail_out = tail_x[global_thread_id] * static_cast<T>(tmp_half_mask) * h2_scale.x;
    if (has_addend) { tmp_tail_out += tail_addend[global_thread_id]; }
    tail_y[global_thread_id] = tmp_tail_out;
  }
}

template<typename T, int pack_size, bool tail, bool has_addend>
__global__ RETURN_VOID_IF_DOUBLE FusedDropoutAddGpu(uint64_t seed, uint64_t offset,
                                                    const int64_t elem_cnt, float rate, float scale,
                                                    int64_t n_tail, const T* x, bool* mask,
                                                    const T* addend, T* y, const T* tail_x,
                                                    bool* tail_mask, const T* tail_addend,
                                                    T* tail_y) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, offset, &state);
  using LoadType = cuda::elementwise::PackType<T, pack_size>;
  using LoadPack = cuda::elementwise::Pack<T, pack_size>;
  using MaskType = cuda::elementwise::PackType<bool, pack_size>;
  using MaskPack = cuda::elementwise::Pack<bool, pack_size>;

  RandPack4 rand_uniform_pack4;
  bool grid_loop_rand_state = 0;

  for (int64_t linear_index = global_thread_id * pack_size; linear_index < elem_cnt;
       linear_index += gridDim.x * blockDim.x * pack_size) {
    if (grid_loop_rand_state == 0) {
      rand_uniform_pack4.storage = hiprand_uniform4(&state);
      grid_loop_rand_state ^= 1;
    } else {
      // Use the last two random numbers we generated in previous iteration.
      rand_uniform_pack4.elem[0] = rand_uniform_pack4.elem[2];
      rand_uniform_pack4.elem[1] = rand_uniform_pack4.elem[3];
      grid_loop_rand_state ^= 1;
    }
    const LoadType* x_load = reinterpret_cast<const LoadType*>(x + linear_index);
    LoadPack x_vec;
    x_vec.storage = *x_load;

    LoadPack addend_vec;
    if (has_addend) {
      const LoadType* addend_load = reinterpret_cast<const LoadType*>(addend + linear_index);
      addend_vec.storage = *addend_load;
    }

    MaskPack mask_vec;
    LoadPack y_vec;
#pragma unroll
    for (int i = 0; i < pack_size; i++) {
      mask_vec.elem[i] = rand_uniform_pack4.elem[i] > rate;
      y_vec.elem[i] = x_vec.elem[i] * mask_vec.elem[i] * scale;
      if (has_addend) { y_vec.elem[i] += addend_vec.elem[i]; }
    }
    *(reinterpret_cast<LoadType*>(y + linear_index)) = y_vec.storage;
    *(reinterpret_cast<MaskType*>(mask + linear_index)) = mask_vec.storage;
  }

  if (tail && global_thread_id < n_tail) {
    const float rand_uniform = hiprand_uniform(&state);
    const bool mask_val = rand_uniform > rate;
    tail_mask[global_thread_id] = mask_val;
    double tmp_tail_out = tail_x[global_thread_id] * mask_val * scale;
    if (has_addend) { tmp_tail_out += tail_addend[global_thread_id]; }
    tail_y[global_thread_id] = tmp_tail_out;
  }
}

unsigned int ComputeGridSize(ep::Stream* stream, const int32_t block_size, const int64_t elem_cnt) {
  auto* hip_stream = stream->As<ep::CudaStream>();
  const int32_t max_threads_multi_process =
      hip_stream->device_properties().maxThreadsPerMultiProcessor;
  const int32_t multi_processor_count = hip_stream->device_properties().multiProcessorCount;
  unsigned int blocks_per_sm = max_threads_multi_process / block_size;
  unsigned int grid_size = std::max((int64_t)1, ((elem_cnt + block_size - 1) / block_size));
  grid_size = std::min((unsigned int)multi_processor_count * blocks_per_sm, grid_size);
  return grid_size;
}

template<typename T, bool has_addend>
void DispatchTail(ep::Stream* stream, const std::shared_ptr<one::CUDAGeneratorImpl>& cuda_generator,
                  const int64_t elem_cnt, float rate, float scale, const T* x, bool* mask,
                  const T* addend, T* y) {
  constexpr int pack_size = GetDropoutPackSize<T>();
  const int64_t pack_num = elem_cnt / pack_size;
  unsigned int grid_size = ComputeGridSize(stream, kBlockSize, pack_num);
  const int64_t tail_offset = pack_num * pack_size;
  const int64_t n_tail = elem_cnt - tail_offset;
  const bool tail = n_tail > 0 ? true : false;
  uint64_t offset = 0;
  uint64_t seed = cuda_generator->current_seed();

  if (tail) {
    // If tail, we need generate randnum one more time, so here we add another `1`.
    uint64_t inc_offset = ((elem_cnt - 1) / (kBlockSize * grid_size * kVecSize) + 1) * kVecSize + 1;
    offset = cuda_generator->get_philox_offset(inc_offset);
    FusedDropoutAddGpu<T, pack_size, true, has_addend>
        <<<grid_size, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            seed, offset, elem_cnt, rate, scale, n_tail, x, mask, addend, y, (x + tail_offset),
            (mask + tail_offset), (addend + tail_offset), (y + tail_offset));
  } else {
    uint64_t inc_offset = ((elem_cnt - 1) / (kBlockSize * grid_size * kVecSize) + 1) * kVecSize;
    offset = cuda_generator->get_philox_offset(inc_offset);
    FusedDropoutAddGpu<T, pack_size, false, has_addend>
        <<<grid_size, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            seed, offset, elem_cnt, rate, scale, n_tail, x, mask, addend, y, nullptr, nullptr,
            nullptr, nullptr);
  }
}

template<typename T>
struct MaskAndScaleFunctor {
  OF_DEVICE_FUNC explicit MaskAndScaleFunctor(float scale) : scale(scale) {}
  __device__ T operator()(T x, bool mask) const {
    return x * static_cast<T>(mask) * static_cast<T>(scale);
  }
  float scale;
};

#if CUDA_VERSION >= 11000
template<>
struct MaskAndScaleFunctor<hip_bfloat16> {
  OF_DEVICE_FUNC explicit MaskAndScaleFunctor(float scale) : scale(scale) {}
  __device__ hip_bfloat16 operator()(hip_bfloat16 x, bool mask) const {
    float float_mask = static_cast<float>(mask);
    return x * static_cast<hip_bfloat16>(float_mask) * static_cast<hip_bfloat16>(scale);
  }
  float scale;
};
#endif

template<typename T>
class DropoutKernelGPU final : public user_op::OpKernel {
 public:
  DropoutKernelGPU() = default;
  ~DropoutKernelGPU() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    const auto& generator = CHECK_JUST(one::MakeGenerator(DeviceType::kCUDA));
    generator->set_current_seed(
        CHECK_JUST(GetOpKernelRandomSeedInCurrentRank(ctx, ctx->Attr<int64_t>("seed"))));
    return std::make_shared<FusedDropoutKernelState>(generator);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    user_op::Tensor* mask = ctx->Tensor4ArgNameAndIndex("mask", 0);
    auto* fused_dropout_kernel_state = dynamic_cast<FusedDropoutKernelState*>(state);
    CHECK_NOTNULL(fused_dropout_kernel_state);
    const auto& generator = fused_dropout_kernel_state->generator();
    CHECK_NOTNULL(generator);
    auto* stream = ctx->stream();
    const auto device_index = stream->device()->device_index();
    std::shared_ptr<one::CUDAGeneratorImpl> cuda_generator =
        CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>(device_index));

    const float rate = ctx->Attr<float>("rate");
    float scale = 0.0;
    if (rate < 1.0f) { scale = 1.0f / (1.0f - rate); }

    if (ctx->has_input("_add_to_output", 0)) {
      const user_op::Tensor* addend = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      DispatchTail<T, true>(
          stream, cuda_generator, in->shape_view().elem_cnt(), rate, scale,
          reinterpret_cast<const T*>(in->dptr()), reinterpret_cast<bool*>(mask->mut_dptr()),
          reinterpret_cast<const T*>(addend->dptr()), reinterpret_cast<T*>(out->mut_dptr()));
    } else {
      DispatchTail<T, false>(stream, cuda_generator, in->shape_view().elem_cnt(), rate, scale,
                             reinterpret_cast<const T*>(in->dptr()),
                             reinterpret_cast<bool*>(mask->mut_dptr()), nullptr,
                             reinterpret_cast<T*>(out->mut_dptr()));
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_DROPOUT_KERNEL_GPU(cpp_type, data_type)                                     \
  REGISTER_USER_KERNEL("dropout").SetCreateFn<DropoutKernelGPU<cpp_type>>().SetIsMatchedHob( \
      (user_op::HobDeviceType() == DeviceType::kCUDA)                                        \
      && (user_op::HobDataType("out", 0) == data_type)                                       \
      && (user_op::HobDataType("mask", 0) == GetDataType<bool>::value))

REGISTER_DROPOUT_KERNEL_GPU(half, DataType::kFloat16);
REGISTER_DROPOUT_KERNEL_GPU(float, DataType::kFloat);
REGISTER_DROPOUT_KERNEL_GPU(double, DataType::kDouble);
#if CUDA_VERSION >= 11000
REGISTER_DROPOUT_KERNEL_GPU(hip_bfloat16, DataType::kBFloat16);
#endif

template<typename T>
class DropoutGradKernelGPU final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  DropoutGradKernelGPU() = default;
  ~DropoutGradKernelGPU() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* mask = ctx->Tensor4ArgNameAndIndex("mask", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const float scale = ctx->Attr<float>("scale");
    const int64_t elem_cnt = dy->shape_view().elem_cnt();
    OF_CUDA_CHECK((cuda::elementwise::Binary(
        MaskAndScaleFunctor<T>(scale), elem_cnt, reinterpret_cast<T*>(dx->mut_dptr()),
        reinterpret_cast<const T*>(dy->dptr()), reinterpret_cast<const bool*>(mask->dptr()),
        ctx->stream()->As<ep::CudaStream>()->hip_stream())));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_DROPOUT_GRAD_KERNEL_GPU(cpp_type, data_type)                                   \
  REGISTER_USER_KERNEL("dropout_grad")                                                          \
      .SetCreateFn<DropoutGradKernelGPU<cpp_type>>()                                            \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                          \
                       && (user_op::HobDataType("dx", 0) == data_type))                         \
      .SetInplaceProposalFn([](const user_op::InferContext&,                                    \
                               user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> { \
        OF_RETURN_IF_ERROR(AddInplaceArgPairFn("dx", 0, "dy", 0, true));                        \
        return Maybe<void>::Ok();                                                               \
      })

REGISTER_DROPOUT_GRAD_KERNEL_GPU(half, DataType::kFloat16);
REGISTER_DROPOUT_GRAD_KERNEL_GPU(float, DataType::kFloat);
REGISTER_DROPOUT_GRAD_KERNEL_GPU(double, DataType::kDouble);
#if CUDA_VERSION >= 11000
REGISTER_DROPOUT_GRAD_KERNEL_GPU(hip_bfloat16, DataType::kBFloat16);
#endif

}  // namespace

}  // namespace oneflow
