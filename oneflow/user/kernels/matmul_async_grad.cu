/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/user/kernels/cublas_fused_mlp_util.cuh"
// CUBLAS_AUX_EPILOGUE only support in cuda11.4 or higher version, in cuda11.4 it need static link.
#if CUDA_VERSION >= 11060

namespace oneflow {

namespace {

class MatmulAsyncGradKernelState final : public user_op::OpKernelState {
 public:
  MatmulAsyncGradKernelState() {
    OF_CUDA_CHECK(hipStreamCreate(&cuda_stream_));
    OF_CUBLAS_CHECK(hipblasLtCreate(&cublas_lt_handle_));
    OF_CUDA_CHECK(hipMalloc(&workspace_, 8 * 1024 * 1024));
  }
  ~MatmulAsyncGradKernelState() {
    OF_CUDA_CHECK(hipStreamSynchronize(cuda_stream_));
    OF_CUBLAS_CHECK(hipblasLtDestroy(cublas_lt_handle_));
    OF_CUDA_CHECK(hipStreamDestroy(cuda_stream_));
    OF_CUDA_CHECK(hipFree(workspace_));
  }
  hipStream_t hip_stream() const { return cuda_stream_; }
  hipblasLtHandle_t cublas_lt_handle() const { return cublas_lt_handle_; }
  size_t cublas_workspace_size() const { return 8 * 1024 * 1024; }
  void* cublas_workspace() const { return workspace_; }

 private:
  hipStream_t cuda_stream_{};
  hipblasLtHandle_t cublas_lt_handle_{};
  void* workspace_{};
};

template<typename T>
class MatmulAsyncGradKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  MatmulAsyncGradKernel() {
    OF_CUDA_CHECK(hipEventCreate(&main_stream_event));
    OF_CUDA_CHECK(hipEventCreate(&async_matmul_grad_event));
  };
  ~MatmulAsyncGradKernel() override {
    OF_CUDA_CHECK(hipEventDestroy(main_stream_event));
    OF_CUDA_CHECK(hipEventDestroy(async_matmul_grad_event));
  };

  std::shared_ptr<user_op::OpKernelCache> InitOpKernelCache(
      user_op::KernelCacheContext* ctx) const override {
    return CreateCublasFusedMLPKernelCache();
  }

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<MatmulAsyncGradKernelState>();
  }

 private:
  hipEvent_t main_stream_event;
  hipEvent_t async_matmul_grad_event;

  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache* cache) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* weight = ctx->Tensor4ArgNameAndIndex("weight", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);

    user_op::Tensor* d_grad = ctx->Tensor4ArgNameAndIndex("d_grad", 0);
    user_op::Tensor* d_weight = ctx->Tensor4ArgNameAndIndex("d_weight", 0);

    const auto* matmul_grad_cache =
        CHECK_NOTNULL(dynamic_cast<const CublasFusedMLPKernelCache*>(cache));
    auto* hip_stream = ctx->stream()->As<ep::CudaStream>();

    auto* kernel_state = dynamic_cast<MatmulAsyncGradKernelState*>(state);

    const DataType data_type = dy->data_type();
    const hipblasComputeType_t cublas_compute_dtype = GetComputeType(data_type);
    const hipDataType cuda_data_type = GetCudaDataType(data_type);
    size_t cublas_m = 0, cublas_n = 0, cublas_k = 0;
    int64_t cublas_lda = 0, cublas_ldb = 0, cublas_ldc = 0;

    double alpha = 1.0;
    auto sp_alpha = GetCublasScalarParameter(alpha, cublas_compute_dtype);
    double beta = 0.0;
    auto sp_beta = GetCublasScalarParameter(beta, cublas_compute_dtype);

    // currently only support 2D matmul.
    DimVector dy_shape(2);
    dy->shape().ToDimVector(&dy_shape);
    DimVector weight_shape(2);
    weight->shape().ToDimVector(&weight_shape);
    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

    InferMatmulCublasMNK(dy_shape, weight_shape,
                         /*transpose_a=*/ep::primitive::BlasTransposeType::N,
                         /*transpose_b=*/ep::primitive::BlasTransposeType::N, &cublas_m, &cublas_n,
                         &cublas_k, &cublas_lda, &cublas_ldb, &cublas_ldc);

    SetCublasAttr(matmul_grad_cache, cublas_compute_dtype, cuda_data_type, /*need_aux=*/false,
                  /*transpose_a=*/ep::primitive::BlasTransposeType::N,
                  /*transpose_b=*/ep::primitive::BlasTransposeType::N, epilogue, nullptr, nullptr,
                  cublas_m, cublas_n, cublas_k, cublas_lda, cublas_ldb, cublas_ldc, 8 * 1024 * 1024);
    /*
    a = dy, b = weight
    cublas_a=weight, cublas_b=dy
    */
    OF_CUDA_CHECK(hipEventRecord(main_stream_event, hip_stream->hip_stream()));
    OF_CUBLAS_CHECK(
        hipblasLtMatmul(hip_stream->cublas_lt_handle(), matmul_grad_cache->operation_desc,
                       &sp_alpha, weight->dptr(), matmul_grad_cache->cublas_a_desc, dy->dptr(),
                       matmul_grad_cache->cublas_b_desc, &sp_beta, d_grad->mut_dptr(),
                       matmul_grad_cache->cublas_c_desc, d_grad->mut_dptr(),
                       matmul_grad_cache->cublas_c_desc, nullptr, hip_stream->cublas_workspace(),
                       hip_stream->cublas_workspace_size(), hip_stream->hip_stream()));

    // currently only support 2D matmul.
    DimVector x_shape(2);
    x->shape().ToDimVector(&x_shape);
    InferMatmulCublasMNK(dy_shape, x_shape,
                         /*transpose_a=*/ep::primitive::BlasTransposeType::T,
                         /*transpose_b=*/ep::primitive::BlasTransposeType::N, &cublas_m, &cublas_n,
                         &cublas_k, &cublas_lda, &cublas_ldb, &cublas_ldc);

    SetCublasAttr(matmul_grad_cache, cublas_compute_dtype, cuda_data_type, /*need_aux=*/false,
                  /*transpose_a=*/ep::primitive::BlasTransposeType::T,
                  /*transpose_b=*/ep::primitive::BlasTransposeType::N, epilogue, nullptr, nullptr,
                  cublas_m, cublas_n, cublas_k, cublas_lda, cublas_ldb, cublas_ldc, 8 * 1024 * 1024);
    OF_CUDA_CHECK(hipStreamWaitEvent(kernel_state->hip_stream(), main_stream_event));
    OF_CUBLAS_CHECK(hipblasLtMatmul(
        kernel_state->cublas_lt_handle(), matmul_grad_cache->operation_desc, &sp_alpha, x->dptr(),
        matmul_grad_cache->cublas_a_desc, dy->dptr(), matmul_grad_cache->cublas_b_desc, &sp_beta,
        d_weight->mut_dptr(), matmul_grad_cache->cublas_c_desc, d_weight->mut_dptr(),
        matmul_grad_cache->cublas_c_desc, nullptr, kernel_state->cublas_workspace(),
        kernel_state->cublas_workspace_size(), kernel_state->hip_stream()));
    OF_CUDA_CHECK(hipEventRecord(async_matmul_grad_event, kernel_state->hip_stream()));
    OF_CUDA_CHECK(hipStreamWaitEvent(hip_stream->hip_stream(), async_matmul_grad_event));
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_MATMUL_ASYNC_GRAD_KERNEL(dtype)                                           \
  REGISTER_USER_KERNEL("matmul_async_grad")                                                \
      .SetCreateFn<MatmulAsyncGradKernel<dtype>>()                                         \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                     \
                       && (user_op::HobDataType("weight", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value));

REGISTER_MATMUL_ASYNC_GRAD_KERNEL(float)
REGISTER_MATMUL_ASYNC_GRAD_KERNEL(double)
REGISTER_MATMUL_ASYNC_GRAD_KERNEL(half)

}  // namespace

}  // namespace oneflow

#endif  // CUDA_VERSION >= 11060
