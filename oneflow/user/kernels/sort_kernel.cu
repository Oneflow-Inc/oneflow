/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/user/kernels/radix_sort.cuh"

namespace oneflow {

template<typename T>
class GpuSortKernel final : public user_op::OpKernel {
 public:
  GpuSortKernel() = default;
  ~GpuSortKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    Memcpy<DeviceType::kGPU>(ctx->device_ctx(), out->mut_dptr<T>(), in->dptr<T>(),
                             in->shape().elem_cnt() * sizeof(T));
    const int32_t instance_size = in->shape().At(in->shape().NumAxes() - 1);
    const int32_t instance_num = in->shape().elem_cnt() / instance_size;
    const std::string& direction = ctx->Attr<std::string>("direction");
    if (direction == "ASCENDING") {
      SortKeysAscending(in->dptr<T>(), instance_num, instance_size, tmp_buffer->mut_dptr<void>(),
                        tmp_buffer->shape().elem_cnt(), out->mut_dptr<T>(),
                        ctx->device_ctx()->hip_stream());
    } else if (direction == "DESCENDING") {
      SortKeysDescending(in->dptr<T>(), instance_num, instance_size, tmp_buffer->mut_dptr<void>(),
                         tmp_buffer->shape().elem_cnt(), out->mut_dptr<T>(),
                         ctx->device_ctx()->hip_stream());
    } else {
      UNIMPLEMENTED();
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GPU_SORT_KERNEL(dtype)                                                     \
  REGISTER_USER_KERNEL("sort")                                                              \
      .SetCreateFn<GpuSortKernel<dtype>>()                                                  \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                                   \
                       & (user_op::HobDataType("out", 0) == GetDataType<dtype>::value))     \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                   \
        const Shape& in_shape = ctx->InputShape("in", 0);                                   \
        const int32_t instance_size = in_shape.dim_vec().back();                            \
        const int32_t instance_num = in_shape.elem_cnt() / instance_size;                   \
        const std::string& direction = ctx->Attr<std::string>("direction");                 \
        if (direction == "ASCENDING") {                                                     \
          return InferTempStorageForSortKeysAscending<dtype>(instance_num, instance_size);  \
        } else if (direction == "DESCENDING") {                                             \
          return InferTempStorageForSortKeysDescending<dtype>(instance_num, instance_size); \
        } else {                                                                            \
          UNIMPLEMENTED();                                                                  \
        }                                                                                   \
      });

REGISTER_GPU_SORT_KERNEL(float)
REGISTER_GPU_SORT_KERNEL(double)
REGISTER_GPU_SORT_KERNEL(int32_t)
REGISTER_GPU_SORT_KERNEL(int64_t)

}  // namespace oneflow
