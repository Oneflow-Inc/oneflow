/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/user/kernels/radix_sort.cuh"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

template<typename T>
class GpuSortKernel final : public user_op::OpKernel {
 public:
  GpuSortKernel() = default;
  ~GpuSortKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    Memcpy<DeviceType::kCUDA>(ctx->stream(), out->mut_dptr<T>(), in->dptr<T>(),
                              in->shape_view().elem_cnt() * sizeof(T));
    const int32_t instance_size = in->shape_view().At(in->shape_view().NumAxes() - 1);
    const int32_t instance_num = in->shape_view().elem_cnt() / instance_size;
    const std::string& direction = ctx->Attr<std::string>("direction");
    if (direction == "ASCENDING") {
      SortKeysAscending(in->dptr<T>(), instance_num, instance_size, tmp_buffer->mut_dptr<void>(),
                        tmp_buffer->shape_view().elem_cnt(), out->mut_dptr<T>(),
                        ctx->stream()->As<ep::CudaStream>()->hip_stream());
    } else if (direction == "DESCENDING") {
      SortKeysDescending(in->dptr<T>(), instance_num, instance_size, tmp_buffer->mut_dptr<void>(),
                         tmp_buffer->shape_view().elem_cnt(), out->mut_dptr<T>(),
                         ctx->stream()->As<ep::CudaStream>()->hip_stream());
    } else {
      UNIMPLEMENTED();
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_SORT_KERNEL(dtype)                                                    \
  REGISTER_USER_KERNEL("sort")                                                              \
      .SetCreateFn<GpuSortKernel<dtype>>()                                                  \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                      \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value))    \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) -> size_t {                         \
        const Shape& in_shape = ctx->InputShape("in", 0);                                   \
        const int32_t instance_size = in_shape.dim_vec().back();                            \
        const int32_t instance_num = in_shape.elem_cnt() / instance_size;                   \
        const std::string& direction = ctx->Attr<std::string>("direction");                 \
        if (direction == "ASCENDING") {                                                     \
          return InferTempStorageForSortKeysAscending<dtype>(instance_num, instance_size);  \
        } else if (direction == "DESCENDING") {                                             \
          return InferTempStorageForSortKeysDescending<dtype>(instance_num, instance_size); \
        } else {                                                                            \
          UNIMPLEMENTED();                                                                  \
          return 0;                                                                         \
        }                                                                                   \
      });

REGISTER_CUDA_SORT_KERNEL(float)
REGISTER_CUDA_SORT_KERNEL(double)
REGISTER_CUDA_SORT_KERNEL(int32_t)
REGISTER_CUDA_SORT_KERNEL(int64_t)

}  // namespace oneflow
