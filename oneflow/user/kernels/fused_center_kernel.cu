#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
struct FusedCenterForwardFunctor {
  __device__ T Compute(T b_x_delta, T b_y_delta) const {
    return (b_x_delta * b_x_delta + b_y_delta * b_y_delta) / static_cast<T>(4.0);
  }
};

template<>
struct FusedCenterForwardFunctor<half> {
  FusedCenterForwardFunctor<float> float_functor;
  __device__ half Compute(half b_x_delta, half b_y_delta) const {
    return __float2half(float_functor.Compute(__half2float(b_x_delta), __half2float(b_y_delta)));
  }
};

template<typename FUNCTOR, typename T>
__global__ void FusedCenterForward(FUNCTOR functor, const int n, const T* b1_x1, const T* b1_x2,
                                   const T* b2_x1, const T* b2_x2, const T* b1_y1, const T* b1_y2,
                                   const T* b2_y1, const T* b2_y2, T* rho) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T b_x_delta = (b2_x1[i] + b2_x2[i] - b1_x1[i] - b1_x2[i]);
    const T b_y_delta = (b2_y1[i] + b2_y2[i] - b1_y1[i] - b1_y2[i]);
    rho[i] = functor.Compute(b_x_delta, b_y_delta);
  }
}

template<typename T>
__global__ void FusedCenterBackward(const int n, const T* b1_x1, const T* b1_x2, const T* b2_x1,
                                    const T* b2_x2, const T* b1_y1, const T* b1_y2, const T* b2_y1,
                                    const T* b2_y2, const T* rho2_diff, T* b1_x1_diff,
                                    T* b1_x2_diff, T* b2_x1_diff, T* b2_x2_diff, T* b1_y1_diff,
                                    T* b1_y2_diff, T* b2_y1_diff, T* b2_y2_diff) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T rho2_diff_i_2 = rho2_diff[i] / static_cast<T>(2.0);

    const T b_x_diff = rho2_diff_i_2 * (b1_x1[i] + b1_x2[i] - b2_x1[i] - b2_x2[i]);
    const T b_y_diff = rho2_diff_i_2 * (b1_y1[i] + b1_y2[i] - b2_y1[i] - b2_y2[i]);

    b1_x1_diff[i] = b_x_diff;
    b1_x2_diff[i] = b_x_diff;
    b2_x1_diff[i] = b_x_diff * static_cast<T>(-1.0);
    b2_x2_diff[i] = b_x_diff * static_cast<T>(-1.0);

    b1_y1_diff[i] = b_y_diff;
    b1_y2_diff[i] = b_y_diff;
    b2_y1_diff[i] = b_y_diff * static_cast<T>(-1.0);
    b2_y2_diff[i] = b_y_diff * static_cast<T>(-1.0);
  }
}

}  // namespace

template<typename T>
class FusedCenterKernel final : public user_op::OpKernel {
 public:
  FusedCenterKernel() = default;
  ~FusedCenterKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* b1_x1 = ctx->Tensor4ArgNameAndIndex("b1_x1", 0);
    const user_op::Tensor* b1_x2 = ctx->Tensor4ArgNameAndIndex("b1_x2", 0);
    const user_op::Tensor* b2_x1 = ctx->Tensor4ArgNameAndIndex("b2_x1", 0);
    const user_op::Tensor* b2_x2 = ctx->Tensor4ArgNameAndIndex("b2_x2", 0);
    const user_op::Tensor* b1_y1 = ctx->Tensor4ArgNameAndIndex("b1_y1", 0);
    const user_op::Tensor* b1_y2 = ctx->Tensor4ArgNameAndIndex("b1_y2", 0);
    const user_op::Tensor* b2_y1 = ctx->Tensor4ArgNameAndIndex("b2_y1", 0);
    const user_op::Tensor* b2_y2 = ctx->Tensor4ArgNameAndIndex("b2_y2", 0);

    user_op::Tensor* rho = ctx->Tensor4ArgNameAndIndex("rho2", 0);

    const int64_t elem_cnt = b1_x1->shape_view().elem_cnt();

    FusedCenterForwardFunctor<T> fused_center_forward_functor{};

    RUN_CUDA_KERNEL((FusedCenterForward<decltype(fused_center_forward_functor), T>), ctx->stream(),
                    elem_cnt, fused_center_forward_functor, elem_cnt, b1_x1->dptr<T>(),
                    b1_x2->dptr<T>(), b2_x1->dptr<T>(), b2_x2->dptr<T>(), b1_y1->dptr<T>(),
                    b1_y2->dptr<T>(), b2_y1->dptr<T>(), b2_y2->dptr<T>(), rho->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_GET_CENTER_DIST_CUDA_KERNEL(dtype)              \
  REGISTER_USER_KERNEL("fused_get_center_dist")                        \
      .SetCreateFn<FusedCenterKernel<dtype>>()                         \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("rho2", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_GET_CENTER_DIST_CUDA_KERNEL(float)
REGISTER_FUSED_GET_CENTER_DIST_CUDA_KERNEL(double)
REGISTER_FUSED_GET_CENTER_DIST_CUDA_KERNEL(half)

template<typename T>
class FusedCenterGradKernel final : public user_op::OpKernel {
 public:
  FusedCenterGradKernel() = default;
  ~FusedCenterGradKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* b1_x1 = ctx->Tensor4ArgNameAndIndex("b1_x1", 0);
    const user_op::Tensor* b1_x2 = ctx->Tensor4ArgNameAndIndex("b1_x2", 0);
    const user_op::Tensor* b2_x1 = ctx->Tensor4ArgNameAndIndex("b2_x1", 0);
    const user_op::Tensor* b2_x2 = ctx->Tensor4ArgNameAndIndex("b2_x2", 0);
    const user_op::Tensor* b1_y1 = ctx->Tensor4ArgNameAndIndex("b1_y1", 0);
    const user_op::Tensor* b1_y2 = ctx->Tensor4ArgNameAndIndex("b1_y2", 0);
    const user_op::Tensor* b2_y1 = ctx->Tensor4ArgNameAndIndex("b2_y1", 0);
    const user_op::Tensor* b2_y2 = ctx->Tensor4ArgNameAndIndex("b2_y2", 0);
    const user_op::Tensor* rho2_diff = ctx->Tensor4ArgNameAndIndex("rho2_diff", 0);

    user_op::Tensor* b1_x1_diff = ctx->Tensor4ArgNameAndIndex("b1_x1_diff", 0);
    user_op::Tensor* b1_x2_diff = ctx->Tensor4ArgNameAndIndex("b1_x2_diff", 0);
    user_op::Tensor* b2_x1_diff = ctx->Tensor4ArgNameAndIndex("b2_x1_diff", 0);
    user_op::Tensor* b2_x2_diff = ctx->Tensor4ArgNameAndIndex("b2_x2_diff", 0);
    user_op::Tensor* b1_y1_diff = ctx->Tensor4ArgNameAndIndex("b1_y1_diff", 0);
    user_op::Tensor* b1_y2_diff = ctx->Tensor4ArgNameAndIndex("b1_y2_diff", 0);
    user_op::Tensor* b2_y1_diff = ctx->Tensor4ArgNameAndIndex("b2_y1_diff", 0);
    user_op::Tensor* b2_y2_diff = ctx->Tensor4ArgNameAndIndex("b2_y2_diff", 0);

    const int64_t elem_cnt = b1_x1_diff->shape_view().elem_cnt();

    RUN_CUDA_KERNEL((FusedCenterBackward<T>), ctx->stream(), elem_cnt, elem_cnt, b1_x1->dptr<T>(),
                    b1_x2->dptr<T>(), b2_x1->dptr<T>(), b2_x2->dptr<T>(), b1_y1->dptr<T>(),
                    b1_y2->dptr<T>(), b2_y1->dptr<T>(), b2_y2->dptr<T>(), rho2_diff->dptr<T>(),
                    b1_x1_diff->mut_dptr<T>(), b1_x2_diff->mut_dptr<T>(), b2_x1_diff->mut_dptr<T>(),
                    b2_x2_diff->mut_dptr<T>(), b1_y1_diff->mut_dptr<T>(), b1_y2_diff->mut_dptr<T>(),
                    b2_y1_diff->mut_dptr<T>(), b2_y2_diff->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_GET_CENTER_DIST_GRAD_CUDA_KERNEL(dtype)         \
  REGISTER_USER_KERNEL("fused_get_center_dist_grad")                   \
      .SetCreateFn<FusedCenterGradKernel<dtype>>()                     \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("b1_x1", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_GET_CENTER_DIST_GRAD_CUDA_KERNEL(float)
REGISTER_FUSED_GET_CENTER_DIST_GRAD_CUDA_KERNEL(double)
REGISTER_FUSED_GET_CENTER_DIST_GRAD_CUDA_KERNEL(half)

}  // namespace oneflow
