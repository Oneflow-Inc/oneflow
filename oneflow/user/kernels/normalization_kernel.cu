
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#ifdef WITH_CUDA

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/kernel/new_kernel_util.h"

namespace oneflow {

namespace {

#if (CUDNN_VERSION >= 7401)
#define BN_ENABLE_EX_API
#endif

void InferDimSizeAndDataFormat(const ShapeView& x_shape, const int32_t axis, int32_t* n, int32_t* c,
                               int32_t* h, int32_t* w, cudnnTensorFormat_t* format) {
  if (x_shape.Count(axis + 1) == 1) {
    if (axis == 0) {
      *n = 1;
      *h = 1;
    } else {
      *n = x_shape.At(0);
      *h = x_shape.Count(1, axis);
    }
    *w = 1;
    *c = x_shape.At(axis);
    *format = CUDNN_TENSOR_NHWC;
  } else {
    *n = x_shape.Count(0, axis);
    *c = x_shape.At(axis);
    *h = x_shape.Count(axis + 1);
    *w = 1;
    *format = CUDNN_TENSOR_NCHW;
  }
}

void InferXYCudnnTensorDesc(const ShapeView& xy_shape, const DataType& data_type,
                            const int32_t axis, cudnnTensorDescriptor_t xy_desc) {
  int32_t n, c, h, w;
  cudnnTensorFormat_t format;
  InferDimSizeAndDataFormat(xy_shape, axis, &n, &c, &h, &w, &format);
  OF_CUDNN_CHECK(
      cudnnSetTensor4dDescriptor(xy_desc, format, GetCudnnDataType(data_type), n, c, h, w));
}

void InferParamCudnnTensorDesc(const cudnnTensorDescriptor_t xy_desc, cudnnBatchNormMode_t mode,
                               cudnnTensorDescriptor_t param_desc) {
  OF_CUDNN_CHECK(cudnnDeriveBNTensorDescriptor(param_desc, xy_desc, mode));
}

class CudnnTensorDescHelper final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(CudnnTensorDescHelper);
  CudnnTensorDescHelper(const ShapeView& xy_shape, const DataType& data_type, const int32_t axis,
                        cudnnBatchNormMode_t mode) {
    OF_CUDNN_CHECK(cudnnCreateTensorDescriptor(&xy_desc_));
    InferXYCudnnTensorDesc(xy_shape, data_type, axis, xy_desc_);
    OF_CUDNN_CHECK(cudnnCreateTensorDescriptor(&param_desc_));
    InferParamCudnnTensorDesc(xy_desc_, mode, param_desc_);
    int n, c, h, w, n_stride, c_stride, h_stride, w_stride;
    OF_CUDNN_CHECK(cudnnGetTensor4dDescriptor(param_desc_, &param_data_type_, &n, &c, &h, &w,
                                              &n_stride, &c_stride, &h_stride, &w_stride));
    param_size_ = c;
  }
  ~CudnnTensorDescHelper() {
    OF_CUDNN_CHECK(cudnnDestroyTensorDescriptor(param_desc_));
    OF_CUDNN_CHECK(cudnnDestroyTensorDescriptor(xy_desc_));
  }

  cudnnTensorDescriptor_t xy_desc() const { return xy_desc_; }

  cudnnTensorDescriptor_t param_desc() const { return param_desc_; }

  void CheckParamTensor(const user_op::Tensor* tensor) const {
    CHECK_EQ(tensor->shape().NumAxes(), 1);
    CHECK_EQ(tensor->shape().At(0), param_size_);
    CHECK_EQ(GetCudnnDataType(tensor->data_type()), param_data_type_);
  }

 private:
  cudnnTensorDescriptor_t xy_desc_ = nullptr;
  cudnnTensorDescriptor_t param_desc_ = nullptr;
  cudnnDataType_t param_data_type_;
  int32_t param_size_ = 0;
};

size_t InferTrainWorkspaceSize(const ShapeView& x_shape, const DataType data_type,
                               const int32_t axis) {
#if defined(BN_ENABLE_EX_API)
  const CudnnTensorDescHelper desc_helper(x_shape, data_type, axis,
                                          CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
  size_t size_in_bytes;
  cudnnHandle_t handle;
  OF_CUDNN_CHECK(cudnnCreate(&handle));
  OF_CUDNN_CHECK(cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
      handle, CUDNN_BATCHNORM_SPATIAL_PERSISTENT, CUDNN_BATCHNORM_OPS_BN, desc_helper.xy_desc(),
      nullptr, desc_helper.xy_desc(), desc_helper.param_desc(), nullptr, &size_in_bytes));
  OF_CUDNN_CHECK(cudnnDestroy(handle));
  return std::max(size_in_bytes, static_cast<size_t>(1));
#else
  return 1;
#endif
}

size_t InferTrainTmpSize(user_op::InferContext* ctx) {
  const auto* x = ctx->TensorDesc4ArgNameAndIndex("x", 0);
  const auto axis = ctx->Attr<int32_t>("axis");
  return InferTrainWorkspaceSize(x->shape(), x->data_type(), axis);
}

size_t InferGradWorkspaceSize(const ShapeView& x_shape, const DataType data_type,
                              const int32_t axis) {
#if defined(BN_ENABLE_EX_API)
  const CudnnTensorDescHelper desc_helper(x_shape, data_type, axis,
                                          CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
  size_t size_in_bytes;
  cudnnHandle_t handle;
  OF_CUDNN_CHECK(cudnnCreate(&handle));
  OF_CUDNN_CHECK(cudnnGetBatchNormalizationBackwardExWorkspaceSize(
      handle, CUDNN_BATCHNORM_SPATIAL_PERSISTENT, CUDNN_BATCHNORM_OPS_BN, desc_helper.xy_desc(),
      nullptr, desc_helper.xy_desc(), nullptr, desc_helper.xy_desc(), desc_helper.param_desc(),
      nullptr, &size_in_bytes));
  OF_CUDNN_CHECK(cudnnDestroy(handle));
  return std::max(size_in_bytes, static_cast<size_t>(1));
#else
  return 1;
#endif
}

size_t InferGradTmpSize(user_op::InferContext* ctx) {
  const auto* dy = ctx->TensorDesc4ArgNameAndIndex("dy", 0);
  const auto axis = ctx->Attr<int32_t>("axis");
  size_t tmp_size = 0;
  if (ctx->user_op_conf().op_type_name() == "normalization_add_relu_grad"
      && !ctx->user_op_conf().has_output("addend_diff", 0)) {
    tmp_size += GetCudaAlignedSize(dy->shape().elem_cnt() * GetSizeOfDataType(dy->data_type()));
  }
  tmp_size += GetCudaAlignedSize(InferGradWorkspaceSize(dy->shape(), dy->data_type(), axis));
  return tmp_size;
}

template<typename T>
class NormalizationInferenceKernel final : public user_op::OpKernel {
 public:
  NormalizationInferenceKernel() = default;
  ~NormalizationInferenceKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const bool training = ctx->Attr<bool>("training");
    CHECK(!training);
    const auto* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    auto* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const auto* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const auto* beta = ctx->Tensor4ArgNameAndIndex("beta", 0);
    auto* moving_mean = ctx->Tensor4ArgNameAndIndex("moving_mean", 0);
    auto* moving_variance = ctx->Tensor4ArgNameAndIndex("moving_variance", 0);
    const auto axis = ctx->Attr<int32_t>("axis");
    const auto epsilon = ctx->Attr<float>("epsilon");

    const DataType data_type = x->data_type();
    CHECK_EQ(x->shape(), y->shape());
    CHECK_EQ(y->data_type(), data_type);
    CHECK_GE(axis, 0);
    CHECK_LT(axis, x->shape().NumAxes());

    const CudnnTensorDescHelper desc_helper(x->shape(), data_type, axis, CUDNN_BATCHNORM_SPATIAL);
    desc_helper.CheckParamTensor(gamma);
    desc_helper.CheckParamTensor(beta);
    desc_helper.CheckParamTensor(moving_mean);
    desc_helper.CheckParamTensor(moving_variance);

    const void* sp_alpha = CudnnSPOnePtr<T>();
    const void* sp_beta;
    if (ctx->user_op_conf().has_input("_add_to_output", 0)) {
      const user_op::Tensor* add_to_output = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      CHECK_EQ(add_to_output->data_type(), y->data_type());
      CHECK_EQ(add_to_output->shape(), y->shape());
      Memcpy<DeviceType::kGPU>(
          ctx->device_ctx(), y->mut_dptr<void>(), add_to_output->dptr<void>(),
          add_to_output->shape().elem_cnt() * GetSizeOfDataType(add_to_output->data_type()));
      sp_beta = CudnnSPOnePtr<T>();
    } else {
      sp_beta = CudnnSPZeroPtr<T>();
    }

    OF_CUDNN_CHECK(cudnnBatchNormalizationForwardInference(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL, sp_alpha, sp_beta,
        desc_helper.xy_desc(), x->dptr(), desc_helper.xy_desc(), y->mut_dptr(),
        desc_helper.param_desc(), gamma->dptr(), beta->dptr(), moving_mean->dptr(),
        moving_variance->dptr(), epsilon));
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_BN_INFERENCE_KERNEL(dtype)                                                     \
  REGISTER_USER_KERNEL("normalization")                                                         \
      .SetCreateFn<NormalizationInferenceKernel<dtype>>()                                       \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                                       \
                       & (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)            \
                       & (user_op::HobAttr<bool>("training") == false))                         \
      .SetInplaceProposalFn([](const user_op::InferContext& ctx,                                \
                               user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> { \
        if (ctx.user_op_conf().has_input("_add_to_output", 0)) {                                \
          OF_RETURN_IF_ERROR(AddInplaceArgPairFn("y", 0, "_add_to_output", 0, true));           \
        }                                                                                       \
        return Maybe<void>::Ok();                                                               \
      });

REGISTER_BN_INFERENCE_KERNEL(float16)
REGISTER_BN_INFERENCE_KERNEL(float)
REGISTER_BN_INFERENCE_KERNEL(double)

#undef REGISTER_BN_INFERENCE_KERNEL

constexpr int64_t kCudaWarpSize = 32;

template<typename T>
__global__ void ReluGpu(int64_t n, const T* x, T* y, int32_t* mask) {
  const int32_t lane_id = threadIdx.x % kCudaWarpSize;
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T x_val = x[i];
    const bool is_positive = (x_val > 0);
    int32_t warp_mask = __ballot_sync(__activemask(), static_cast<int>(is_positive));
    if (lane_id == 0) { mask[i / kCudaWarpSize] = warp_mask; }
    y[i] = is_positive ? x_val : 0;
  }
}

template<>
__global__ void ReluGpu<half>(int64_t n, const half* x, half* y, int32_t* mask) {
  const int32_t lane_id = threadIdx.x % kCudaWarpSize;
  const half zero = __float2half(0.0f);
  CUDA_1D_KERNEL_LOOP(i, n) {
    const half x_val = x[i];
    const bool is_positive = __hgt(x_val, zero);
    int32_t warp_mask = __ballot_sync(__activemask(), static_cast<int>(is_positive));
    if (lane_id == 0) { mask[i / kCudaWarpSize] = warp_mask; }
    y[i] = is_positive ? x_val : zero;
  }
}

template<typename T>
__global__ void AddReluGpu(int64_t n, const T* x, const T* addend, T* y, int32_t* mask) {
  const int32_t lane_id = threadIdx.x % kCudaWarpSize;
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T sum = x[i] + addend[i];
    const bool is_positive = (sum > 0);
    int32_t warp_mask = __ballot_sync(__activemask(), static_cast<int>(is_positive));
    if (lane_id == 0) { mask[i / kCudaWarpSize] = warp_mask; }
    y[i] = is_positive ? sum : 0;
  }
}

template<>
__global__ void AddReluGpu<half>(int64_t n, const half* x, const half* addend, half* y,
                                 int32_t* mask) {
  const int32_t lane_id = threadIdx.x % kCudaWarpSize;
  const half zero = __float2half(0.0f);
  CUDA_1D_KERNEL_LOOP(i, n) {
    const half sum = __hadd(x[i], addend[i]);
    const bool is_positive = __hgt(sum, zero);
    int32_t warp_mask = __ballot_sync(__activemask(), static_cast<int>(is_positive));
    if (lane_id == 0) { mask[i / kCudaWarpSize] = warp_mask; }
    y[i] = is_positive ? sum : zero;
  }
}

template<typename T>
void Relu(DeviceCtx* device_ctx, int64_t n, const T* x, T* y, int32_t* mask) {
  ReluGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, device_ctx->cuda_stream()>>>(
      n, x, y, mask);
}

template<>
void Relu<float16>(DeviceCtx* device_ctx, int64_t n, const float16* x, float16* y, int32_t* mask) {
  Relu<half>(device_ctx, n, reinterpret_cast<const half*>(x), reinterpret_cast<half*>(y), mask);
}

template<typename T>
void AddRelu(DeviceCtx* device_ctx, int64_t n, const T* x, const T* addend, T* y, int32_t* mask) {
  AddReluGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, device_ctx->cuda_stream()>>>(
      n, x, addend, y, mask);
}

template<>
void AddRelu<float16>(DeviceCtx* device_ctx, int64_t n, const float16* x, const float16* addend,
                      float16* y, int32_t* mask) {
  AddRelu<half>(device_ctx, n, reinterpret_cast<const half*>(x),
                reinterpret_cast<const half*>(addend), reinterpret_cast<half*>(y), mask);
}

template<typename T>
__global__ void ReluBackwardGpu(int64_t n, const int32_t* mask, const T* dy, T* addend_diff) {
  int32_t lane_id = threadIdx.x % kCudaWarpSize;
  CUDA_1D_KERNEL_LOOP(i, n) {
    int32_t mask_val = mask[i / kCudaWarpSize];
    bool is_positive = mask_val & (1 << lane_id);
    addend_diff[i] = static_cast<T>(is_positive) * dy[i];
  }
}

template<typename T>
void ReluBackward(DeviceCtx* device_ctx, int64_t n, const int32_t* mask, const T* dy,
                  T* addend_diff) {
  ReluBackwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, device_ctx->cuda_stream()>>>(
          n, mask, dy, addend_diff);
}

template<>
void ReluBackward<float16>(DeviceCtx* device_ctx, int64_t n, const int32_t* mask, const float16* dy,
                           float16* addend_diff) {
  ReluBackward<half>(device_ctx, n, mask, reinterpret_cast<const half*>(dy),
                     reinterpret_cast<half*>(addend_diff));
}

template<typename T>
class NormalizationTrainKernel final : public user_op::OpKernel {
 public:
  NormalizationTrainKernel() = default;
  ~NormalizationTrainKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    if (ctx->user_op_conf().op_type_name() == "normalization") {
      CHECK(ctx->Attr<bool>("training"));
    }
    const auto* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    auto* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const auto* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const auto* beta = ctx->Tensor4ArgNameAndIndex("beta", 0);
    auto* moving_mean = ctx->Tensor4ArgNameAndIndex("moving_mean", 0);
    auto* moving_variance = ctx->Tensor4ArgNameAndIndex("moving_variance", 0);
    const auto axis = ctx->Attr<int32_t>("axis");
    const auto epsilon = ctx->Attr<float>("epsilon");
    const auto momentum = ctx->Attr<float>("momentum");
    auto* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    auto* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);

    const DataType data_type = x->data_type();
    CHECK_EQ(x->shape(), y->shape());
    CHECK_EQ(y->data_type(), data_type);
    CHECK_GE(axis, 0);
    CHECK_LT(axis, x->shape().NumAxes());

    const CudnnTensorDescHelper desc_helper(x->shape(), data_type, axis,
                                            CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
    desc_helper.CheckParamTensor(gamma);
    desc_helper.CheckParamTensor(beta);
    desc_helper.CheckParamTensor(moving_mean);
    desc_helper.CheckParamTensor(moving_variance);
    desc_helper.CheckParamTensor(mean);
    desc_helper.CheckParamTensor(inv_variance);

    const void* sp_alpha = CudnnSPOnePtr<T>();
    const void* sp_beta;
    if (ctx->user_op_conf().has_input("_add_to_output", 0)) {
      const user_op::Tensor* add_to_output = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      CHECK_EQ(add_to_output->data_type(), y->data_type());
      CHECK_EQ(add_to_output->shape(), y->shape());
      Memcpy<DeviceType::kGPU>(
          ctx->device_ctx(), y->mut_dptr<void>(), add_to_output->dptr<void>(),
          add_to_output->shape().elem_cnt() * GetSizeOfDataType(add_to_output->data_type()));
      sp_beta = CudnnSPOnePtr<T>();
    } else {
      sp_beta = CudnnSPZeroPtr<T>();
    }

#if defined(BN_ENABLE_EX_API)
    size_t workspace_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
        CUDNN_BATCHNORM_OPS_BN, desc_helper.xy_desc(), nullptr, desc_helper.xy_desc(),
        desc_helper.param_desc(), nullptr, &workspace_size));
    size_t reserve_space_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
        CUDNN_BATCHNORM_OPS_BN, nullptr, desc_helper.xy_desc(), &reserve_space_size));
    auto* workspace = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    if (reserve_space_size == 0 && workspace_size <= workspace->shape().elem_cnt()) {
      OF_CUDNN_CHECK(cudnnBatchNormalizationForwardTrainingEx(
          ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
          CUDNN_BATCHNORM_OPS_BN, sp_alpha, sp_beta, desc_helper.xy_desc(), x->dptr(), nullptr,
          nullptr, desc_helper.xy_desc(), y->mut_dptr(), desc_helper.param_desc(), gamma->dptr(),
          beta->dptr(), 1.0 - momentum, moving_mean->mut_dptr(), moving_variance->mut_dptr(),
          epsilon, mean->mut_dptr(), inv_variance->mut_dptr(), nullptr, workspace->mut_dptr(),
          workspace->shape().elem_cnt(), nullptr, 0));
    } else {
      OF_CUDNN_CHECK(cudnnBatchNormalizationForwardTraining(
          ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT, sp_alpha, sp_beta,
          desc_helper.xy_desc(), x->dptr(), desc_helper.xy_desc(), y->mut_dptr(),
          desc_helper.param_desc(), gamma->dptr(), beta->dptr(), 1.0 - momentum,
          moving_mean->mut_dptr(), moving_variance->mut_dptr(), epsilon, mean->mut_dptr(),
          inv_variance->mut_dptr()));
    }
#else
    OF_CUDNN_CHECK(cudnnBatchNormalizationForwardTraining(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT, sp_alpha, sp_beta,
        desc_helper.xy_desc(), x->dptr(), desc_helper.xy_desc(), y->mut_dptr(),
        desc_helper.param_desc(), gamma->dptr(), beta->dptr(), 1.0 - momentum,
        moving_mean->mut_dptr(), moving_variance->mut_dptr(), epsilon, mean->mut_dptr(),
        inv_variance->mut_dptr()));
#endif

    if (ctx->user_op_conf().op_type_name() == "normalization_add_relu") {
      CHECK(!ctx->user_op_conf().has_input("_add_to_output", 0));
      const int64_t elem_cnt = x->shape().elem_cnt();
      auto* mask = ctx->Tensor4ArgNameAndIndex("reserve_space", 0);
      if (ctx->user_op_conf().has_input("addend", 0)) {
        const auto* addend = ctx->Tensor4ArgNameAndIndex("addend", 0);
        AddRelu(ctx->device_ctx(), elem_cnt, y->dptr<T>(), addend->dptr<T>(), y->mut_dptr<T>(),
                mask->mut_dptr<int32_t>());
      } else {
        Relu(ctx->device_ctx(), elem_cnt, y->dptr<T>(), y->mut_dptr<T>(),
             mask->mut_dptr<int32_t>());
      }
    }
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_BN_TRAIN_KERNEL(dtype)                                                         \
  REGISTER_USER_KERNEL("normalization")                                                         \
      .SetCreateFn<NormalizationTrainKernel<dtype>>()                                           \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                                       \
                       & (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)            \
                       & (user_op::HobAttr<bool>("training") == true))                          \
      .SetInferTmpSizeFn(InferTrainTmpSize)                                                     \
      .SetInplaceProposalFn([](const user_op::InferContext& ctx,                                \
                               user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> { \
        if (ctx.user_op_conf().has_input("_add_to_output", 0)) {                                \
          OF_RETURN_IF_ERROR(AddInplaceArgPairFn("y", 0, "_add_to_output", 0, true));           \
        }                                                                                       \
        return Maybe<void>::Ok();                                                               \
      });

REGISTER_BN_TRAIN_KERNEL(float16)
REGISTER_BN_TRAIN_KERNEL(float)
REGISTER_BN_TRAIN_KERNEL(double)

#define REGISTER_BN_ADD_RELU_KERNEL(dtype)                                            \
  REGISTER_USER_KERNEL("normalization_add_relu")                                      \
      .SetCreateFn<NormalizationTrainKernel<dtype>>()                                 \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                             \
                       & (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn(InferTrainTmpSize);

REGISTER_BN_ADD_RELU_KERNEL(float16)
REGISTER_BN_ADD_RELU_KERNEL(float)
REGISTER_BN_ADD_RELU_KERNEL(double)

template<typename T>
class NormalizationGradUserKernel final : public user_op::OpKernel {
 public:
  NormalizationGradUserKernel() = default;
  ~NormalizationGradUserKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    auto* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const auto* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const auto* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    auto* gamma_diff = ctx->Tensor4ArgNameAndIndex("gamma_diff", 0);
    auto* beta_diff = ctx->Tensor4ArgNameAndIndex("beta_diff", 0);
    const auto* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const auto* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    auto* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const auto axis = ctx->Attr<int32_t>("axis");
    const auto epsilon = ctx->Attr<float>("epsilon");

    const DataType data_type = x->data_type();
    CHECK_EQ(dy->shape(), x->shape());
    CHECK_EQ(dy->data_type(), data_type);
    CHECK_EQ(dx->shape(), x->shape());
    CHECK_EQ(dx->data_type(), data_type);
    CHECK_GE(axis, 0);
    CHECK_LT(axis, x->shape().NumAxes());

    const CudnnTensorDescHelper desc_helper(x->shape(), data_type, axis,
                                            CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
    desc_helper.CheckParamTensor(gamma);
    desc_helper.CheckParamTensor(gamma_diff);
    desc_helper.CheckParamTensor(beta_diff);
    desc_helper.CheckParamTensor(mean);
    desc_helper.CheckParamTensor(inv_variance);

    void* bn_workspace_ptr;
    size_t bn_workspace_size;
    const void* bn_dy_ptr;

    if (ctx->user_op_conf().op_type_name() == "normalization_grad") {
      bn_workspace_ptr = tmp_buffer->mut_dptr();
      bn_workspace_size = tmp_buffer->shape().elem_cnt();
      bn_dy_ptr = dy->dptr();
    } else if (ctx->user_op_conf().op_type_name() == "normalization_add_relu_grad") {
      const int64_t elem_cnt = dy->shape().elem_cnt();
      const auto* mask = ctx->Tensor4ArgNameAndIndex("reserve_space", 0);
      user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
      if (ctx->user_op_conf().has_output("addend_diff", 0)) {
        user_op::Tensor* addend_diff = ctx->Tensor4ArgNameAndIndex("addend_diff", 0);
        ReluBackward(ctx->device_ctx(), elem_cnt, mask->dptr<int32_t>(), dy->dptr<T>(),
                     addend_diff->mut_dptr<T>());
        bn_workspace_ptr = tmp_buffer->mut_dptr();
        bn_workspace_size = tmp_buffer->shape().elem_cnt();
        bn_dy_ptr = addend_diff->dptr();
      } else {
        const size_t tmp_buffer_size = tmp_buffer->shape().elem_cnt();
        const size_t relu_dx_size =
            GetCudaAlignedSize(dy->shape().elem_cnt() * GetSizeOfDataType(dy->data_type()));
        CHECK_GE(tmp_buffer_size, relu_dx_size);
        ReluBackward(ctx->device_ctx(), elem_cnt, mask->dptr<int32_t>(), dy->dptr<T>(),
                     reinterpret_cast<T*>(tmp_buffer->mut_dptr()));
        bn_workspace_ptr = tmp_buffer->mut_dptr<char>() + relu_dx_size;
        bn_workspace_size = tmp_buffer_size - relu_dx_size;
        bn_dy_ptr = tmp_buffer->dptr();
      }
    } else {
      UNIMPLEMENTED();
    }

#if defined(BN_ENABLE_EX_API)
    size_t workspace_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationBackwardExWorkspaceSize(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
        CUDNN_BATCHNORM_OPS_BN, desc_helper.xy_desc(), nullptr, desc_helper.xy_desc(), nullptr,
        desc_helper.xy_desc(), desc_helper.param_desc(), nullptr, &workspace_size));
    size_t reserve_space_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
        CUDNN_BATCHNORM_OPS_BN, nullptr, desc_helper.xy_desc(), &reserve_space_size));
    if (reserve_space_size == 0 && workspace_size <= bn_workspace_size) {
      OF_CUDNN_CHECK(cudnnBatchNormalizationBackwardEx(
          ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
          CUDNN_BATCHNORM_OPS_BN, CudnnSPOnePtr<T>(), CudnnSPZeroPtr<T>(), CudnnSPOnePtr<T>(),
          CudnnSPZeroPtr<T>(), desc_helper.xy_desc(), x->dptr(), nullptr, nullptr,
          desc_helper.xy_desc(), bn_dy_ptr, nullptr, nullptr, desc_helper.xy_desc(), dx->mut_dptr(),
          desc_helper.param_desc(), gamma->dptr(), nullptr, gamma_diff->mut_dptr(),
          beta_diff->mut_dptr(), epsilon, mean->dptr(), inv_variance->dptr(), nullptr,
          bn_workspace_ptr, bn_workspace_size, nullptr, 0));
    } else {
      OF_CUDNN_CHECK(cudnnBatchNormalizationBackward(
          ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT, CudnnSPOnePtr<T>(),
          CudnnSPZeroPtr<T>(), CudnnSPOnePtr<T>(), CudnnSPZeroPtr<T>(), desc_helper.xy_desc(),
          x->dptr(), desc_helper.xy_desc(), bn_dy_ptr, desc_helper.xy_desc(), dx->mut_dptr(),
          desc_helper.param_desc(), gamma->dptr(), gamma_diff->mut_dptr(), beta_diff->mut_dptr(),
          epsilon, mean->dptr(), inv_variance->dptr()));
    }
#else
    OF_CUDNN_CHECK(cudnnBatchNormalizationBackward(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT, CudnnSPOnePtr<T>(),
        CudnnSPZeroPtr<T>(), CudnnSPOnePtr<T>(), CudnnSPZeroPtr<T>(), desc_helper.xy_desc(),
        x->dptr(), desc_helper.xy_desc(), bn_dy_ptr, desc_helper.xy_desc(), dx->mut_dptr(),
        desc_helper.param_desc(), gamma->dptr(), gamma_diff->mut_dptr(), beta_diff->mut_dptr(),
        epsilon, mean->dptr(), inv_variance->dptr()));
#endif
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_BN_GRAD_KERNEL(dtype)                                                 \
  REGISTER_USER_KERNEL("normalization_grad")                                           \
      .SetCreateFn<NormalizationGradUserKernel<dtype>>()                               \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                              \
                       & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn(InferGradTmpSize);

REGISTER_BN_GRAD_KERNEL(float16)
REGISTER_BN_GRAD_KERNEL(float)
REGISTER_BN_GRAD_KERNEL(double)

#define REGISTER_BN_ADD_RELU_GRAD_KERNEL(dtype)                                        \
  REGISTER_USER_KERNEL("normalization_add_relu_grad")                                  \
      .SetCreateFn<NormalizationGradUserKernel<dtype>>()                               \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                              \
                       & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn(InferGradTmpSize);

REGISTER_BN_ADD_RELU_GRAD_KERNEL(float16)
REGISTER_BN_ADD_RELU_GRAD_KERNEL(float)
REGISTER_BN_ADD_RELU_GRAD_KERNEL(double)

#if (CUDNN_VERSION >= 7401)

size_t InferFusedNormalizationAddReluTmpSize(user_op::InferContext* ctx) {
  const auto* x = ctx->TensorDesc4ArgNameAndIndex("x", 0);
  const auto axis = ctx->Attr<int32_t>("axis");
  const CudnnTensorDescHelper desc_helper(x->shape(), x->data_type(), axis,
                                          CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
  size_t size_in_bytes;
  cudnnHandle_t handle;
  OF_CUDNN_CHECK(cudnnCreate(&handle));
  CudnnActivationDesc activation_desc(CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0);
  cudnnBatchNormOps_t ops;
  cudnnTensorDescriptor_t z_desc;
  if (ctx->user_op_conf().has_input("addend", 0)) {
    ops = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
    z_desc = desc_helper.xy_desc();
  } else {
    ops = CUDNN_BATCHNORM_OPS_BN_ACTIVATION;
    z_desc = nullptr;
  }
  OF_CUDNN_CHECK(cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
      handle, CUDNN_BATCHNORM_SPATIAL_PERSISTENT, ops, desc_helper.xy_desc(), z_desc,
      desc_helper.xy_desc(), desc_helper.param_desc(), activation_desc.Get(), &size_in_bytes));
  OF_CUDNN_CHECK(cudnnDestroy(handle));
  return std::max(size_in_bytes, static_cast<size_t>(1));
}

size_t InferFusedNormalizationAddReluGradTmpSize(user_op::InferContext* ctx) {
  const auto* x = ctx->TensorDesc4ArgNameAndIndex("x", 0);
  const auto axis = ctx->Attr<int32_t>("axis");
  const CudnnTensorDescHelper desc_helper(x->shape(), x->data_type(), axis,
                                          CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
  size_t size_in_bytes;
  cudnnHandle_t handle;
  OF_CUDNN_CHECK(cudnnCreate(&handle));
  CudnnActivationDesc activation_desc(CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0);
  cudnnBatchNormOps_t ops;
  cudnnTensorDescriptor_t z_desc;
  if (ctx->user_op_conf().has_output("addend_diff", 0)) {
    ops = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
    z_desc = desc_helper.xy_desc();
  } else {
    ops = CUDNN_BATCHNORM_OPS_BN_ACTIVATION;
    z_desc = nullptr;
  }
  OF_CUDNN_CHECK(cudnnGetBatchNormalizationBackwardExWorkspaceSize(
      handle, CUDNN_BATCHNORM_SPATIAL_PERSISTENT, ops, desc_helper.xy_desc(), desc_helper.xy_desc(),
      desc_helper.xy_desc(), z_desc, desc_helper.xy_desc(), desc_helper.param_desc(),
      activation_desc.Get(), &size_in_bytes));
  OF_CUDNN_CHECK(cudnnDestroy(handle));
  return std::max(size_in_bytes, static_cast<size_t>(1));
}

template<typename T>
class FusedNormalizationAddReluKernel final : public user_op::OpKernel {
 public:
  FusedNormalizationAddReluKernel() = default;
  ~FusedNormalizationAddReluKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    auto* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const auto* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const auto* beta = ctx->Tensor4ArgNameAndIndex("beta", 0);
    auto* moving_mean = ctx->Tensor4ArgNameAndIndex("moving_mean", 0);
    auto* moving_variance = ctx->Tensor4ArgNameAndIndex("moving_variance", 0);
    const auto axis = ctx->Attr<int32_t>("axis");
    const auto epsilon = ctx->Attr<float>("epsilon");
    const auto momentum = ctx->Attr<float>("momentum");
    auto* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    auto* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    auto* reserve_space = ctx->Tensor4ArgNameAndIndex("reserve_space", 0);
    auto* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    const DataType data_type = x->data_type();
    CHECK_EQ(x->shape(), y->shape());
    CHECK_EQ(y->data_type(), data_type);
    CHECK_GE(axis, 0);
    CHECK_LT(axis, x->shape().NumAxes());

    const CudnnTensorDescHelper desc_helper(x->shape(), data_type, axis,
                                            CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
    desc_helper.CheckParamTensor(gamma);
    desc_helper.CheckParamTensor(beta);
    desc_helper.CheckParamTensor(moving_mean);
    desc_helper.CheckParamTensor(moving_variance);
    desc_helper.CheckParamTensor(mean);
    desc_helper.CheckParamTensor(inv_variance);

    CudnnActivationDesc activation_desc(CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0);
    cudnnTensorDescriptor_t z_desc;
    const void* z_ptr;
    cudnnBatchNormOps_t ops;
    if (ctx->user_op_conf().has_input("addend", 0)) {
      z_desc = desc_helper.xy_desc();
      z_ptr = ctx->Tensor4ArgNameAndIndex("addend", 0)->dptr();
      ops = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
    } else {
      z_desc = nullptr;
      z_ptr = nullptr;
      ops = CUDNN_BATCHNORM_OPS_BN_ACTIVATION;
    }

    size_t min_workspace_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT, ops,
        desc_helper.xy_desc(), z_desc, desc_helper.xy_desc(), desc_helper.param_desc(),
        activation_desc.Get(), &min_workspace_size));
    const size_t workspace_size = tmp_buffer->shape().elem_cnt();
    CHECK_GE(workspace_size, min_workspace_size);
    size_t min_reserve_space_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT, ops,
        activation_desc.Get(), desc_helper.xy_desc(), &min_reserve_space_size));
    const size_t reserve_space_size = reserve_space->shape().elem_cnt();
    CHECK_GE(reserve_space_size, min_reserve_space_size);

    OF_CUDNN_CHECK(cudnnBatchNormalizationForwardTrainingEx(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT, ops,
        CudnnSPOnePtr<T>(), CudnnSPZeroPtr<T>(), desc_helper.xy_desc(), x->dptr(), z_desc, z_ptr,
        desc_helper.xy_desc(), y->mut_dptr(), desc_helper.param_desc(), gamma->dptr(), beta->dptr(),
        1.0 - momentum, moving_mean->mut_dptr(), moving_variance->mut_dptr(), epsilon,
        mean->mut_dptr(), inv_variance->mut_dptr(), activation_desc.Get(), tmp_buffer->mut_dptr(),
        workspace_size, reserve_space->mut_dptr(), reserve_space_size));
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_BN_ADD_RELU_KERNEL(dtype)                                      \
  REGISTER_USER_KERNEL("cudnn_fused_normalization_add_relu")                          \
      .SetCreateFn<FusedNormalizationAddReluKernel<dtype>>()                          \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                             \
                       & (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn(InferFusedNormalizationAddReluTmpSize);

REGISTER_FUSED_BN_ADD_RELU_KERNEL(float16)

template<typename T>
class FusedNormalizationAddReluGradUserKernel final : public user_op::OpKernel {
 public:
  FusedNormalizationAddReluGradUserKernel() = default;
  ~FusedNormalizationAddReluGradUserKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const auto* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    auto* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const auto* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const auto* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const auto* beta = ctx->Tensor4ArgNameAndIndex("beta", 0);
    auto* gamma_diff = ctx->Tensor4ArgNameAndIndex("gamma_diff", 0);
    auto* beta_diff = ctx->Tensor4ArgNameAndIndex("beta_diff", 0);
    const auto* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const auto* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    const auto* reserve_space = ctx->Tensor4ArgNameAndIndex("reserve_space", 0);
    auto* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const auto axis = ctx->Attr<int32_t>("axis");
    const auto epsilon = ctx->Attr<float>("epsilon");

    const DataType data_type = x->data_type();
    CHECK_EQ(dy->shape(), x->shape());
    CHECK_EQ(dy->data_type(), data_type);
    CHECK_EQ(dx->shape(), x->shape());
    CHECK_EQ(dx->data_type(), data_type);
    CHECK_GE(axis, 0);
    CHECK_LT(axis, x->shape().NumAxes());

    const CudnnTensorDescHelper desc_helper(x->shape(), data_type, axis,
                                            CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
    desc_helper.CheckParamTensor(gamma);
    desc_helper.CheckParamTensor(beta);
    desc_helper.CheckParamTensor(gamma_diff);
    desc_helper.CheckParamTensor(beta_diff);
    desc_helper.CheckParamTensor(mean);
    desc_helper.CheckParamTensor(inv_variance);

    CudnnActivationDesc activation_desc(CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0);
    cudnnTensorDescriptor_t dz_desc;
    void* dz_ptr;
    cudnnBatchNormOps_t ops;
    if (ctx->user_op_conf().has_output("addend_diff", 0)) {
      dz_desc = desc_helper.xy_desc();
      dz_ptr = ctx->Tensor4ArgNameAndIndex("addend_diff", 0)->mut_dptr();
      ops = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
    } else {
      dz_desc = nullptr;
      dz_ptr = nullptr;
      ops = CUDNN_BATCHNORM_OPS_BN_ACTIVATION;
    }

    size_t min_workspace_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationBackwardExWorkspaceSize(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT, ops,
        desc_helper.xy_desc(), desc_helper.xy_desc(), desc_helper.xy_desc(), dz_desc,
        desc_helper.xy_desc(), desc_helper.param_desc(), activation_desc.Get(),
        &min_workspace_size));
    const size_t workspace_size = tmp_buffer->shape().elem_cnt();
    CHECK_GE(workspace_size, min_workspace_size);
    size_t min_reserve_space_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT, ops,
        activation_desc.Get(), desc_helper.xy_desc(), &min_reserve_space_size));
    const size_t reserve_space_size = reserve_space->shape().elem_cnt();
    CHECK_GE(reserve_space_size, min_reserve_space_size);
    OF_CUDNN_CHECK(cudnnBatchNormalizationBackwardEx(
        ctx->device_ctx()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT, ops,
        CudnnSPOnePtr<T>(), CudnnSPZeroPtr<T>(), CudnnSPOnePtr<T>(), CudnnSPZeroPtr<T>(),
        desc_helper.xy_desc(), x->dptr(), desc_helper.xy_desc(), y->dptr(), desc_helper.xy_desc(),
        dy->dptr(), dz_desc, dz_ptr, desc_helper.xy_desc(), dx->mut_dptr(),
        desc_helper.param_desc(), gamma->dptr(), beta->dptr(), gamma_diff->mut_dptr(),
        beta_diff->mut_dptr(), epsilon, mean->dptr(), inv_variance->dptr(), activation_desc.Get(),
        tmp_buffer->mut_dptr(), workspace_size, const_cast<void*>(reserve_space->dptr()),
        reserve_space_size));
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_BN_ADD_RELU_GRAD_KERNEL(dtype)                                  \
  REGISTER_USER_KERNEL("cudnn_fused_normalization_add_relu_grad")                      \
      .SetCreateFn<FusedNormalizationAddReluGradUserKernel<dtype>>()                   \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                              \
                       & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn(InferFusedNormalizationAddReluGradTmpSize);

REGISTER_FUSED_BN_ADD_RELU_GRAD_KERNEL(float16)

#endif

}  // namespace
}  // namespace oneflow

#endif
