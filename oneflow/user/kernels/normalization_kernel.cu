
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#ifdef WITH_CUDA

#include <unordered_map>

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/ep/cuda/cuda_stream.h"
#include "oneflow/core/device/cuda_pseudo_bfloat16.h"
#if CUDA_VERSION >= 11000
#include <cuda_bf16.h>
#endif  // CUDA_VERSION >= 11000
#include <cudnn.h>

#if (CUDNN_VERSION >= 7401)
#define BN_ENABLE_EX_API
#endif

namespace oneflow {

namespace {

cudnnBatchNormMode_t getCudnnBatchNormMode(const int64_t dim) {
  if (dim == 2) {
    return CUDNN_BATCHNORM_PER_ACTIVATION;
  } else if (ParseBooleanFromEnv("ONEFLOW_ENABLE_NHWC", false)) {
    return CUDNN_BATCHNORM_SPATIAL_PERSISTENT;
  } else {
    // NOTE(Liang Depeng): The new CUDNN_BATCHNORM_SPATIAL_PERSISTENT mode was
    // introduced in CuDNN 7 for performance optimization, but it results in
    // accuracy losses in convolution models such as ResNeXt-101 and
    // video R(2+1)D. We will fall back to the normal CUDNN_BATCHNORM_SPATIAL
    return CUDNN_BATCHNORM_SPATIAL;
  }
}

void InferDimSizeAndDataFormat(const ShapeView& x_shape, const int32_t axis, int32_t* n, int32_t* c,
                               int32_t* h, int32_t* w, cudnnTensorFormat_t* format) {
  if (x_shape.Count(axis + 1) == 1) {
    if (axis == 0) {
      *n = 1;
      *h = 1;
    } else {
      *n = x_shape.At(0);
      *h = x_shape.Count(1, axis);
    }
    *w = 1;
    *c = x_shape.At(axis);
    *format = CUDNN_TENSOR_NHWC;
  } else {
    *n = x_shape.Count(0, axis);
    *c = x_shape.At(axis);
    *h = x_shape.Count(axis + 1);
    *w = 1;
    *format = CUDNN_TENSOR_NCHW;
  }
}

void InferXYCudnnTensorDesc(const ShapeView& xy_shape, const DataType& data_type,
                            const int32_t axis, cudnnTensorDescriptor_t xy_desc) {
  int32_t n, c, h, w;
  cudnnTensorFormat_t format;
  InferDimSizeAndDataFormat(xy_shape, axis, &n, &c, &h, &w, &format);
  OF_CUDNN_CHECK(
      cudnnSetTensor4dDescriptor(xy_desc, format, GetCudnnDataType(data_type), n, c, h, w));
}

void InferParamCudnnTensorDesc(const cudnnTensorDescriptor_t xy_desc, cudnnBatchNormMode_t mode,
                               cudnnTensorDescriptor_t param_desc) {
  OF_CUDNN_CHECK(cudnnDeriveBNTensorDescriptor(param_desc, xy_desc, mode));
}

class CudnnTensorDescHelper final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(CudnnTensorDescHelper);
  CudnnTensorDescHelper(const ShapeView& xy_shape, const DataType& data_type, const int32_t axis,
                        cudnnBatchNormMode_t mode) {
    OF_CUDNN_CHECK(cudnnCreateTensorDescriptor(&xy_desc_));
    InferXYCudnnTensorDesc(xy_shape, data_type, axis, xy_desc_);
    OF_CUDNN_CHECK(cudnnCreateTensorDescriptor(&param_desc_));
    InferParamCudnnTensorDesc(xy_desc_, mode, param_desc_);
    int n, c, h, w, n_stride, c_stride, h_stride, w_stride;
    OF_CUDNN_CHECK(cudnnGetTensor4dDescriptor(param_desc_, &param_data_type_, &n, &c, &h, &w,
                                              &n_stride, &c_stride, &h_stride, &w_stride));
    param_size_ = c;
  }
  ~CudnnTensorDescHelper() {
    OF_CUDNN_CHECK(cudnnDestroyTensorDescriptor(param_desc_));
    OF_CUDNN_CHECK(cudnnDestroyTensorDescriptor(xy_desc_));
  }

  cudnnTensorDescriptor_t xy_desc() const { return xy_desc_; }

  cudnnTensorDescriptor_t param_desc() const { return param_desc_; }

  void CheckParamTensor(const user_op::Tensor* tensor) const {
    CHECK_NOTNULL(tensor);
    CHECK_EQ(tensor->shape_view().NumAxes(), 1);
    CHECK_EQ(tensor->shape_view().At(0), param_size_);
    CHECK_EQ(GetCudnnDataType(tensor->data_type()), param_data_type_);
  }

 private:
  cudnnTensorDescriptor_t xy_desc_ = nullptr;
  cudnnTensorDescriptor_t param_desc_ = nullptr;
  cudnnDataType_t param_data_type_;
  int32_t param_size_ = 0;
};

size_t InferTrainWorkspaceSize(const ShapeView& x_shape, const DataType data_type,
                               const int32_t axis) {
#if defined(BN_ENABLE_EX_API)
  cudnnBatchNormMode_t mode = getCudnnBatchNormMode(x_shape.NumAxes());
  const CudnnTensorDescHelper desc_helper(x_shape, data_type, axis, mode);
  size_t size_in_bytes;
  cudnnHandle_t handle = Singleton<CudnnHandlePool>::Get()->Get();
  OF_CUDNN_CHECK(cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
      handle, mode, CUDNN_BATCHNORM_OPS_BN, desc_helper.xy_desc(), nullptr, desc_helper.xy_desc(),
      desc_helper.param_desc(), nullptr, &size_in_bytes));
  Singleton<CudnnHandlePool>::Get()->Put(handle);
  return std::max(size_in_bytes, static_cast<size_t>(1));
#else
  return 1;
#endif
}

size_t InferTrainTmpSize(user_op::InferContext* ctx) {
  const auto& x = ctx->InputTensorDesc("x", 0);
  const auto axis = ctx->Attr<int32_t>("axis");
  return InferTrainWorkspaceSize(x.shape(), x.data_type(), axis);
}

size_t InferGradWorkspaceSize(const ShapeView& x_shape, const DataType data_type,
                              const int32_t axis) {
#if defined(BN_ENABLE_EX_API)
  cudnnBatchNormMode_t mode = getCudnnBatchNormMode(x_shape.NumAxes());
  const CudnnTensorDescHelper desc_helper(x_shape, data_type, axis, mode);
  size_t size_in_bytes;
  cudnnHandle_t handle = Singleton<CudnnHandlePool>::Get()->Get();
  OF_CUDNN_CHECK(cudnnGetBatchNormalizationBackwardExWorkspaceSize(
      handle, mode, CUDNN_BATCHNORM_OPS_BN, desc_helper.xy_desc(), nullptr, desc_helper.xy_desc(),
      nullptr, desc_helper.xy_desc(), desc_helper.param_desc(), nullptr, &size_in_bytes));
  Singleton<CudnnHandlePool>::Get()->Put(handle);
  return std::max(size_in_bytes, static_cast<size_t>(1));
#else
  return 1;
#endif
}

size_t InferGradTmpSize(user_op::InferContext* ctx) {
  const auto& dy = ctx->InputTensorDesc("dy", 0);
  const auto axis = ctx->Attr<int32_t>("axis");
  size_t tmp_size = 0;
  if (ctx->op_type_name() == "normalization_add_relu_grad" && !ctx->has_output("addend_diff", 0)) {
    tmp_size += GetCudaAlignedSize(dy.shape().elem_cnt() * GetSizeOfDataType(dy.data_type()));
  }
  tmp_size += GetCudaAlignedSize(InferGradWorkspaceSize(dy.shape(), dy.data_type(), axis));
  return tmp_size;
}

class NormalizationInferenceKernel final : public user_op::OpKernel,
                                           public user_op::CudaGraphSupport {
 public:
  NormalizationInferenceKernel() = default;
  ~NormalizationInferenceKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const bool training = ctx->Attr<bool>("training");
    CHECK(!training);
    const auto* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    auto* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const auto* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const auto* beta = ctx->Tensor4ArgNameAndIndex("beta", 0);
    auto* moving_mean = ctx->Tensor4ArgNameAndIndex("moving_mean", 0);
    auto* moving_variance = ctx->Tensor4ArgNameAndIndex("moving_variance", 0);
    const auto axis = ctx->Attr<int32_t>("axis");
    const auto epsilon = ctx->Attr<float>("epsilon");

    const DataType data_type = x->data_type();
    CHECK_EQ(x->shape_view(), y->shape_view());
    CHECK_EQ(y->data_type(), data_type);
    CHECK_GE(axis, 0);
    CHECK_LT(axis, x->shape_view().NumAxes());

    cudnnBatchNormMode_t mode = getCudnnBatchNormMode(x->shape_view().NumAxes());
    const CudnnTensorDescHelper desc_helper(x->shape_view(), data_type, axis, mode);
    desc_helper.CheckParamTensor(gamma);
    desc_helper.CheckParamTensor(beta);
    desc_helper.CheckParamTensor(moving_mean);
    desc_helper.CheckParamTensor(moving_variance);

    const void* sp_alpha = CudnnSPOnePtr(data_type);
    const void* sp_beta;
    if (ctx->has_input("_add_to_output", 0)) {
      const user_op::Tensor* add_to_output = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      CHECK_EQ(add_to_output->data_type(), y->data_type());
      CHECK_EQ(add_to_output->shape_view(), y->shape_view());
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), y->mut_dptr<void>(), add_to_output->dptr<void>(),
          add_to_output->shape_view().elem_cnt() * GetSizeOfDataType(add_to_output->data_type()));
      sp_beta = CudnnSPOnePtr(data_type);
    } else {
      sp_beta = CudnnSPZeroPtr(data_type);
    }

    OF_CUDNN_CHECK(cudnnBatchNormalizationForwardInference(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), mode, sp_alpha, sp_beta,
        desc_helper.xy_desc(), x->dptr(), desc_helper.xy_desc(), y->mut_dptr(),
        desc_helper.param_desc(), gamma->dptr(), beta->dptr(), moving_mean->dptr(),
        moving_variance->dptr(), epsilon));
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("normalization")
    .SetCreateFn<NormalizationInferenceKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)
                     && (user_op::HobAttr<bool>("training") == false))
    .SetInplaceProposalFn([](const user_op::InferContext& ctx,
                             user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> {
      if (ctx.has_input("_add_to_output", 0)) {
        OF_RETURN_IF_ERROR(AddInplaceArgPairFn("y", 0, "_add_to_output", 0, true));
      }
      return Maybe<void>::Ok();
    });

constexpr int64_t kCudaWarpSize = 32;

template<typename T>
__global__ void ReluGpu(int64_t n, const T* x, T* y, int32_t* mask) {
  const int32_t lane_id = threadIdx.x % kCudaWarpSize;
  const T zero = static_cast<T>(0.f);
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T x_val = x[i];
    const bool is_positive = (x_val > zero);
    int32_t warp_mask = __ballot_sync(__activemask(), static_cast<int>(is_positive));
    if (lane_id == 0) { mask[i / kCudaWarpSize] = warp_mask; }
    y[i] = is_positive ? x_val : zero;
  }
}

template<typename T>
__global__ void AddReluGpu(int64_t n, const T* x, const T* addend, T* y, int32_t* mask) {
  const int32_t lane_id = threadIdx.x % kCudaWarpSize;
  const T zero = static_cast<T>(0.f);
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T sum = x[i] + addend[i];
    const bool is_positive = (sum > zero);
    int32_t warp_mask = __ballot_sync(__activemask(), static_cast<int>(is_positive));
    if (lane_id == 0) { mask[i / kCudaWarpSize] = warp_mask; }
    y[i] = is_positive ? sum : zero;
  }
}

template<typename T>
void Relu(ep::Stream* stream, int64_t n, const T* x, T* y, int32_t* mask) {
  ReluGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
               stream->As<ep::CudaStream>()->cuda_stream()>>>(n, x, y, mask);
}

template<typename T>
void AddRelu(ep::Stream* stream, int64_t n, const T* x, const T* addend, T* y, int32_t* mask) {
  AddReluGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                  stream->As<ep::CudaStream>()->cuda_stream()>>>(n, x, addend, y, mask);
}

template<typename T>
__global__ void ReluBackwardGpu(int64_t n, const int32_t* mask, const T* dy, T* addend_diff) {
  int32_t lane_id = threadIdx.x % kCudaWarpSize;
  CUDA_1D_KERNEL_LOOP(i, n) {
    int32_t mask_val = mask[i / kCudaWarpSize];
    bool is_positive = mask_val & (1 << lane_id);
    addend_diff[i] = static_cast<T>(is_positive) * dy[i];
  }
}

#if CUDA_VERSION >= 11000

template<>
__global__ void ReluBackwardGpu<nv_bfloat16>(int64_t n, const int32_t* mask, const nv_bfloat16* dy,
                                             nv_bfloat16* addend_diff) {
  int32_t lane_id = threadIdx.x % kCudaWarpSize;
  CUDA_1D_KERNEL_LOOP(i, n) {
    int32_t mask_val = mask[i / kCudaWarpSize];
    bool is_positive = mask_val & (1 << lane_id);
    addend_diff[i] = static_cast<nv_bfloat16>(static_cast<float>(is_positive)) * dy[i];
  }
}

#endif

template<typename T>
void ReluBackward(ep::Stream* stream, int64_t n, const int32_t* mask, const T* dy, T* addend_diff) {
  ReluBackwardGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                       stream->As<ep::CudaStream>()->cuda_stream()>>>(n, mask, dy, addend_diff);
}

void Relu(ep::Stream* stream, int64_t n, const DataType data_type, const void* x, void* y,
          int32_t* mask) {
  if (data_type == kFloat) {
    Relu<float>(stream, n, reinterpret_cast<const float*>(x), reinterpret_cast<float*>(y), mask);
  } else if (data_type == kDouble) {
    Relu<double>(stream, n, reinterpret_cast<const double*>(x), reinterpret_cast<double*>(y), mask);
  } else if (data_type == kFloat16) {
    Relu<half>(stream, n, reinterpret_cast<const half*>(x), reinterpret_cast<half*>(y), mask);
  } else if (data_type == kBFloat16) {
#if CUDA_VERSION >= 11000
    Relu<nv_bfloat16>(stream, n, reinterpret_cast<const nv_bfloat16*>(x),
                      reinterpret_cast<nv_bfloat16*>(y), mask);
#else
    UNIMPLEMENTED();
#endif
  } else {
    UNIMPLEMENTED();
  }
}
void AddRelu(ep::Stream* stream, int64_t n, const DataType data_type, const void* x,
             const void* addend, void* y, int32_t* mask) {
  if (data_type == kFloat) {
    AddRelu<float>(stream, n, reinterpret_cast<const float*>(x),
                   reinterpret_cast<const float*>(addend), reinterpret_cast<float*>(y), mask);
  } else if (data_type == kDouble) {
    AddRelu<double>(stream, n, reinterpret_cast<const double*>(x),
                    reinterpret_cast<const double*>(addend), reinterpret_cast<double*>(y), mask);
  } else if (data_type == kFloat16) {
    AddRelu<half>(stream, n, reinterpret_cast<const half*>(x),
                  reinterpret_cast<const half*>(addend), reinterpret_cast<half*>(y), mask);
  } else if (data_type == kBFloat16) {
#if CUDA_VERSION >= 11000
    AddRelu<nv_bfloat16>(stream, n, reinterpret_cast<const nv_bfloat16*>(x),
                         reinterpret_cast<const nv_bfloat16*>(addend),
                         reinterpret_cast<nv_bfloat16*>(y), mask);
#else
    UNIMPLEMENTED();
#endif
  } else {
    UNIMPLEMENTED();
  }
}
void ReluBackward(ep::Stream* stream, int64_t n, const DataType data_type, const int32_t* mask,
                  const void* dy, void* addend_diff) {
  if (data_type == kFloat) {
    ReluBackward<float>(stream, n, mask, reinterpret_cast<const float*>(dy),
                        reinterpret_cast<float*>(addend_diff));
  } else if (data_type == kDouble) {
    ReluBackward<double>(stream, n, mask, reinterpret_cast<const double*>(dy),
                         reinterpret_cast<double*>(addend_diff));
  } else if (data_type == kFloat16) {
    ReluBackward<half>(stream, n, mask, reinterpret_cast<const half*>(dy),
                       reinterpret_cast<half*>(addend_diff));
  } else if (data_type == kBFloat16) {
#if CUDA_VERSION >= 11000
    ReluBackward<nv_bfloat16>(stream, n, mask, reinterpret_cast<const nv_bfloat16*>(dy),
                              reinterpret_cast<nv_bfloat16*>(addend_diff));
#else
    UNIMPLEMENTED();
#endif
  } else {
    UNIMPLEMENTED();
  }
}

class NormalizationTrainKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  NormalizationTrainKernel() = default;
  ~NormalizationTrainKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    if (ctx->op_type_name() == "normalization") { CHECK(ctx->Attr<bool>("training")); }
    const auto* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    auto* y = ctx->Tensor4ArgNameAndIndex("y", 0);

    const auto axis = ctx->Attr<int32_t>("axis");
    const auto epsilon = ctx->Attr<float>("epsilon");
    const auto momentum = ctx->Attr<float>("momentum");

    const DataType data_type = x->data_type();
    CHECK_EQ(x->shape_view(), y->shape_view());
    CHECK_EQ(y->data_type(), data_type);
    CHECK_GE(axis, 0);
    CHECK_LT(axis, x->shape_view().NumAxes());
    cudnnBatchNormMode_t mode = getCudnnBatchNormMode(x->shape_view().NumAxes());
    const CudnnTensorDescHelper desc_helper(x->shape_view(), data_type, axis, mode);

    const auto* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const auto* beta = ctx->Tensor4ArgNameAndIndex("beta", 0);
    auto* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    auto* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    desc_helper.CheckParamTensor(gamma);
    desc_helper.CheckParamTensor(beta);
    desc_helper.CheckParamTensor(mean);
    desc_helper.CheckParamTensor(inv_variance);

    user_op::Tensor* moving_mean = nullptr;
    user_op::Tensor* moving_variance = nullptr;
    if (ctx->has_input("moving_mean", 0)) {
      CHECK(ctx->has_input("moving_variance", 0));
      moving_mean = ctx->Tensor4ArgNameAndIndex("moving_mean", 0);
      moving_variance = ctx->Tensor4ArgNameAndIndex("moving_variance", 0);
      desc_helper.CheckParamTensor(moving_mean);
      desc_helper.CheckParamTensor(moving_variance);
    }

    const void* sp_alpha = CudnnSPOnePtr(data_type);
    const void* sp_beta;
    if (ctx->has_input("_add_to_output", 0)) {
      const user_op::Tensor* add_to_output = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      CHECK_EQ(add_to_output->data_type(), y->data_type());
      CHECK_EQ(add_to_output->shape_view(), y->shape_view());
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), y->mut_dptr<void>(), add_to_output->dptr<void>(),
          add_to_output->shape_view().elem_cnt() * GetSizeOfDataType(add_to_output->data_type()));
      sp_beta = CudnnSPOnePtr(data_type);
    } else {
      sp_beta = CudnnSPZeroPtr(data_type);
    }

#if defined(BN_ENABLE_EX_API)
    size_t workspace_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), mode, CUDNN_BATCHNORM_OPS_BN,
        desc_helper.xy_desc(), nullptr, desc_helper.xy_desc(), desc_helper.param_desc(), nullptr,
        &workspace_size));
    size_t reserve_space_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), mode, CUDNN_BATCHNORM_OPS_BN, nullptr,
        desc_helper.xy_desc(), &reserve_space_size));
    auto* workspace = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    if (reserve_space_size == 0 && workspace_size <= workspace->shape_view().elem_cnt()) {
      OF_CUDNN_CHECK(cudnnBatchNormalizationForwardTrainingEx(
          ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), mode, CUDNN_BATCHNORM_OPS_BN,
          sp_alpha, sp_beta, desc_helper.xy_desc(), x->dptr(), nullptr, nullptr,
          desc_helper.xy_desc(), y->mut_dptr(), desc_helper.param_desc(), gamma->dptr(),
          beta->dptr(), 1.0 - momentum, moving_mean ? moving_mean->mut_dptr() : NULL,
          moving_variance ? moving_variance->mut_dptr() : NULL, epsilon, mean->mut_dptr(),
          inv_variance->mut_dptr(), nullptr, workspace->mut_dptr(),
          workspace->shape_view().elem_cnt(), nullptr, 0));
    } else {
      OF_CUDNN_CHECK(cudnnBatchNormalizationForwardTraining(
          ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), mode, sp_alpha, sp_beta,
          desc_helper.xy_desc(), x->dptr(), desc_helper.xy_desc(), y->mut_dptr(),
          desc_helper.param_desc(), gamma->dptr(), beta->dptr(), 1.0 - momentum,
          moving_mean ? moving_mean->mut_dptr() : NULL,
          moving_variance ? moving_variance->mut_dptr() : NULL, epsilon, mean->mut_dptr(),
          inv_variance->mut_dptr()));
    }
#else
    OF_CUDNN_CHECK(cudnnBatchNormalizationForwardTraining(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), mode, sp_alpha, sp_beta,
        desc_helper.xy_desc(), x->dptr(), desc_helper.xy_desc(), y->mut_dptr(),
        desc_helper.param_desc(), gamma->dptr(), beta->dptr(), 1.0 - momentum,
        moving_mean ? moving_mean->mut_dptr() : NULL,
        moving_variance ? moving_variance->mut_dptr() : NULL, epsilon, mean->mut_dptr(),
        inv_variance->mut_dptr()));
#endif

    if (ctx->op_type_name() == "normalization_add_relu") {
      CHECK(!ctx->has_input("_add_to_output", 0));
      const int64_t elem_cnt = x->shape_view().elem_cnt();
      auto* mask = ctx->Tensor4ArgNameAndIndex("reserve_space", 0);
      if (ctx->has_input("addend", 0)) {
        const auto* addend = ctx->Tensor4ArgNameAndIndex("addend", 0);
        AddRelu(ctx->stream(), elem_cnt, data_type, y->dptr(), addend->dptr(), y->mut_dptr(),
                mask->mut_dptr<int32_t>());
      } else {
        Relu(ctx->stream(), elem_cnt, data_type, y->dptr(), y->mut_dptr(),
             mask->mut_dptr<int32_t>());
      }
    }
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("normalization")
    .SetCreateFn<NormalizationTrainKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)
                     && (user_op::HobAttr<bool>("training") == true))
    .SetInferTmpSizeFn(InferTrainTmpSize)
    .SetInplaceProposalFn([](const user_op::InferContext& ctx,
                             user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> {
      if (ctx.has_input("_add_to_output", 0)) {
        OF_RETURN_IF_ERROR(AddInplaceArgPairFn("y", 0, "_add_to_output", 0, true));
      }
      return Maybe<void>::Ok();
    });

REGISTER_USER_KERNEL("normalization_add_relu")
    .SetCreateFn<NormalizationTrainKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA))
    .SetInferTmpSizeFn(InferTrainTmpSize);

class NormalizationGradUserKernel final : public user_op::OpKernel,
                                          public user_op::CudaGraphSupport {
 public:
  NormalizationGradUserKernel() = default;
  ~NormalizationGradUserKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    auto* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const auto* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const auto* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    auto* gamma_diff = ctx->Tensor4ArgNameAndIndex("gamma_diff", 0);
    auto* beta_diff = ctx->Tensor4ArgNameAndIndex("beta_diff", 0);
    const auto* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const auto* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    auto* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const auto axis = ctx->Attr<int32_t>("axis");
    const auto epsilon = ctx->Attr<float>("epsilon");

    const DataType data_type = x->data_type();
    CHECK_EQ(dy->shape_view(), x->shape_view());
    CHECK_EQ(dy->data_type(), data_type);
    CHECK_EQ(dx->shape_view(), x->shape_view());
    CHECK_EQ(dx->data_type(), data_type);
    CHECK_GE(axis, 0);
    CHECK_LT(axis, x->shape_view().NumAxes());
    cudnnBatchNormMode_t mode = getCudnnBatchNormMode(x->shape_view().NumAxes());
    const CudnnTensorDescHelper desc_helper(x->shape_view(), data_type, axis, mode);
    desc_helper.CheckParamTensor(gamma);
    desc_helper.CheckParamTensor(gamma_diff);
    desc_helper.CheckParamTensor(beta_diff);
    desc_helper.CheckParamTensor(mean);
    desc_helper.CheckParamTensor(inv_variance);

    void* bn_workspace_ptr;
    size_t bn_workspace_size;
    const void* bn_dy_ptr;

    if (ctx->op_type_name() == "normalization_grad") {
      bn_workspace_ptr = tmp_buffer->mut_dptr();
      bn_workspace_size = tmp_buffer->shape_view().elem_cnt();
      bn_dy_ptr = dy->dptr();
    } else if (ctx->op_type_name() == "normalization_add_relu_grad") {
      const int64_t elem_cnt = dy->shape_view().elem_cnt();
      const auto* mask = ctx->Tensor4ArgNameAndIndex("reserve_space", 0);
      user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
      if (ctx->has_output("addend_diff", 0)) {
        user_op::Tensor* addend_diff = ctx->Tensor4ArgNameAndIndex("addend_diff", 0);
        ReluBackward(ctx->stream(), elem_cnt, data_type, mask->dptr<int32_t>(), dy->dptr(),
                     addend_diff->mut_dptr());
        bn_workspace_ptr = tmp_buffer->mut_dptr();
        bn_workspace_size = tmp_buffer->shape_view().elem_cnt();
        bn_dy_ptr = addend_diff->dptr();
      } else {
        const size_t tmp_buffer_size = tmp_buffer->shape_view().elem_cnt();
        const size_t relu_dx_size =
            GetCudaAlignedSize(dy->shape_view().elem_cnt() * GetSizeOfDataType(data_type));
        CHECK_GE(tmp_buffer_size, relu_dx_size);
        ReluBackward(ctx->stream(), elem_cnt, data_type, mask->dptr<int32_t>(), dy->dptr(),
                     tmp_buffer->mut_dptr());
        bn_workspace_ptr = tmp_buffer->mut_dptr<char>() + relu_dx_size;
        bn_workspace_size = tmp_buffer_size - relu_dx_size;
        bn_dy_ptr = tmp_buffer->dptr();
      }
    } else {
      UNIMPLEMENTED();
    }

#if defined(BN_ENABLE_EX_API)
    size_t workspace_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationBackwardExWorkspaceSize(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), mode, CUDNN_BATCHNORM_OPS_BN,
        desc_helper.xy_desc(), nullptr, desc_helper.xy_desc(), nullptr, desc_helper.xy_desc(),
        desc_helper.param_desc(), nullptr, &workspace_size));
    size_t reserve_space_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), mode, CUDNN_BATCHNORM_OPS_BN, nullptr,
        desc_helper.xy_desc(), &reserve_space_size));
    if (reserve_space_size == 0 && workspace_size <= bn_workspace_size) {
      OF_CUDNN_CHECK(cudnnBatchNormalizationBackwardEx(
          ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), mode, CUDNN_BATCHNORM_OPS_BN,
          CudnnSPOnePtr(data_type), CudnnSPZeroPtr(data_type), CudnnSPOnePtr(data_type),
          CudnnSPZeroPtr(data_type), desc_helper.xy_desc(), x->dptr(), nullptr, nullptr,
          desc_helper.xy_desc(), bn_dy_ptr, nullptr, nullptr, desc_helper.xy_desc(), dx->mut_dptr(),
          desc_helper.param_desc(), gamma->dptr(), nullptr, gamma_diff->mut_dptr(),
          beta_diff->mut_dptr(), epsilon, mean->dptr(), inv_variance->dptr(), nullptr,
          bn_workspace_ptr, bn_workspace_size, nullptr, 0));
    } else {
      OF_CUDNN_CHECK(cudnnBatchNormalizationBackward(
          ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), mode, CudnnSPOnePtr(data_type),
          CudnnSPZeroPtr(data_type), CudnnSPOnePtr(data_type), CudnnSPZeroPtr(data_type),
          desc_helper.xy_desc(), x->dptr(), desc_helper.xy_desc(), bn_dy_ptr, desc_helper.xy_desc(),
          dx->mut_dptr(), desc_helper.param_desc(), gamma->dptr(), gamma_diff->mut_dptr(),
          beta_diff->mut_dptr(), epsilon, mean->dptr(), inv_variance->dptr()));
    }
#else
    OF_CUDNN_CHECK(cudnnBatchNormalizationBackward(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), mode, CudnnSPOnePtr(data_type),
        CudnnSPZeroPtr(data_type), CudnnSPOnePtr(data_type), CudnnSPZeroPtr(data_type),
        desc_helper.xy_desc(), x->dptr(), desc_helper.xy_desc(), bn_dy_ptr, desc_helper.xy_desc(),
        dx->mut_dptr(), desc_helper.param_desc(), gamma->dptr(), gamma_diff->mut_dptr(),
        beta_diff->mut_dptr(), epsilon, mean->dptr(), inv_variance->dptr()));
#endif
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("normalization_grad")
    .SetCreateFn<NormalizationGradUserKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA))
    .SetInferTmpSizeFn(InferGradTmpSize);

#define REGISTER_BN_ADD_RELU_GRAD_KERNEL(dtype)
REGISTER_USER_KERNEL("normalization_add_relu_grad")
    .SetCreateFn<NormalizationGradUserKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA))
    .SetInferTmpSizeFn(InferGradTmpSize);

#if (CUDNN_VERSION >= 7401)

size_t InferFusedNormalizationAddReluTmpSize(user_op::InferContext* ctx) {
  const auto& x = ctx->InputTensorDesc("x", 0);
  const auto axis = ctx->Attr<int32_t>("axis");
  const CudnnTensorDescHelper desc_helper(x.shape(), x.data_type(), axis,
                                          CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
  size_t size_in_bytes;
  cudnnHandle_t handle = Singleton<CudnnHandlePool>::Get()->Get();
  CudnnActivationDesc activation_desc(CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0);
  cudnnBatchNormOps_t ops;
  cudnnTensorDescriptor_t z_desc;
  if (ctx->has_input("addend", 0)) {
    ops = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
    z_desc = desc_helper.xy_desc();
  } else {
    ops = CUDNN_BATCHNORM_OPS_BN_ACTIVATION;
    z_desc = nullptr;
  }
  OF_CUDNN_CHECK(cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
      handle, CUDNN_BATCHNORM_SPATIAL_PERSISTENT, ops, desc_helper.xy_desc(), z_desc,
      desc_helper.xy_desc(), desc_helper.param_desc(), activation_desc.Get(), &size_in_bytes));
  Singleton<CudnnHandlePool>::Get()->Put(handle);
  return std::max(size_in_bytes, static_cast<size_t>(1));
}

size_t InferFusedNormalizationAddReluGradTmpSize(user_op::InferContext* ctx) {
  const auto& x = ctx->InputTensorDesc("x", 0);
  const auto axis = ctx->Attr<int32_t>("axis");
  const CudnnTensorDescHelper desc_helper(x.shape(), x.data_type(), axis,
                                          CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
  size_t size_in_bytes;
  cudnnHandle_t handle = Singleton<CudnnHandlePool>::Get()->Get();
  CudnnActivationDesc activation_desc(CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0);
  cudnnBatchNormOps_t ops;
  cudnnTensorDescriptor_t z_desc;
  if (ctx->has_output("addend_diff", 0)) {
    ops = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
    z_desc = desc_helper.xy_desc();
  } else {
    ops = CUDNN_BATCHNORM_OPS_BN_ACTIVATION;
    z_desc = nullptr;
  }
  OF_CUDNN_CHECK(cudnnGetBatchNormalizationBackwardExWorkspaceSize(
      handle, CUDNN_BATCHNORM_SPATIAL_PERSISTENT, ops, desc_helper.xy_desc(), desc_helper.xy_desc(),
      desc_helper.xy_desc(), z_desc, desc_helper.xy_desc(), desc_helper.param_desc(),
      activation_desc.Get(), &size_in_bytes));
  Singleton<CudnnHandlePool>::Get()->Put(handle);
  return std::max(size_in_bytes, static_cast<size_t>(1));
}

class FusedNormalizationAddReluKernel final : public user_op::OpKernel,
                                              public user_op::CudaGraphSupport {
 public:
  FusedNormalizationAddReluKernel() = default;
  ~FusedNormalizationAddReluKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    auto* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const auto* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const auto* beta = ctx->Tensor4ArgNameAndIndex("beta", 0);
    auto* moving_mean = ctx->Tensor4ArgNameAndIndex("moving_mean", 0);
    auto* moving_variance = ctx->Tensor4ArgNameAndIndex("moving_variance", 0);
    const auto axis = ctx->Attr<int32_t>("axis");
    const auto epsilon = ctx->Attr<float>("epsilon");
    const auto momentum = ctx->Attr<float>("momentum");
    auto* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    auto* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    auto* reserve_space = ctx->Tensor4ArgNameAndIndex("reserve_space", 0);
    auto* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    const DataType data_type = x->data_type();
    CHECK_EQ(x->shape_view(), y->shape_view());
    CHECK_EQ(y->data_type(), data_type);
    CHECK_GE(axis, 0);
    CHECK_LT(axis, x->shape_view().NumAxes());

    const CudnnTensorDescHelper desc_helper(x->shape_view(), data_type, axis,
                                            CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
    desc_helper.CheckParamTensor(gamma);
    desc_helper.CheckParamTensor(beta);
    desc_helper.CheckParamTensor(moving_mean);
    desc_helper.CheckParamTensor(moving_variance);
    desc_helper.CheckParamTensor(mean);
    desc_helper.CheckParamTensor(inv_variance);

    CudnnActivationDesc activation_desc(CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0);
    cudnnTensorDescriptor_t z_desc;
    const void* z_ptr;
    cudnnBatchNormOps_t ops;
    if (ctx->has_input("addend", 0)) {
      z_desc = desc_helper.xy_desc();
      z_ptr = ctx->Tensor4ArgNameAndIndex("addend", 0)->dptr();
      ops = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
    } else {
      z_desc = nullptr;
      z_ptr = nullptr;
      ops = CUDNN_BATCHNORM_OPS_BN_ACTIVATION;
    }

    size_t min_workspace_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
        ops, desc_helper.xy_desc(), z_desc, desc_helper.xy_desc(), desc_helper.param_desc(),
        activation_desc.Get(), &min_workspace_size));
    const size_t workspace_size = tmp_buffer->shape_view().elem_cnt();
    CHECK_GE(workspace_size, min_workspace_size);
    size_t min_reserve_space_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
        ops, activation_desc.Get(), desc_helper.xy_desc(), &min_reserve_space_size));
    const size_t reserve_space_size = reserve_space->shape_view().elem_cnt();
    CHECK_GE(reserve_space_size, min_reserve_space_size);

    OF_CUDNN_CHECK(cudnnBatchNormalizationForwardTrainingEx(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
        ops, CudnnSPOnePtr(data_type), CudnnSPZeroPtr(data_type), desc_helper.xy_desc(), x->dptr(),
        z_desc, z_ptr, desc_helper.xy_desc(), y->mut_dptr(), desc_helper.param_desc(),
        gamma->dptr(), beta->dptr(), 1.0 - momentum, moving_mean->mut_dptr(),
        moving_variance->mut_dptr(), epsilon, mean->mut_dptr(), inv_variance->mut_dptr(),
        activation_desc.Get(), tmp_buffer->mut_dptr(), workspace_size, reserve_space->mut_dptr(),
        reserve_space_size));
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("cudnn_fused_normalization_add_relu")
    .SetCreateFn<FusedNormalizationAddReluKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA))
    .SetInferTmpSizeFn(InferFusedNormalizationAddReluTmpSize);

class FusedNormalizationAddReluGradUserKernel final : public user_op::OpKernel,
                                                      public user_op::CudaGraphSupport {
 public:
  FusedNormalizationAddReluGradUserKernel() = default;
  ~FusedNormalizationAddReluGradUserKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const auto* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    auto* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const auto* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const auto* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const auto* beta = ctx->Tensor4ArgNameAndIndex("beta", 0);
    auto* gamma_diff = ctx->Tensor4ArgNameAndIndex("gamma_diff", 0);
    auto* beta_diff = ctx->Tensor4ArgNameAndIndex("beta_diff", 0);
    const auto* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const auto* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    const auto* reserve_space = ctx->Tensor4ArgNameAndIndex("reserve_space", 0);
    auto* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const auto axis = ctx->Attr<int32_t>("axis");
    const auto epsilon = ctx->Attr<float>("epsilon");

    const DataType data_type = x->data_type();
    CHECK_EQ(dy->shape_view(), x->shape_view());
    CHECK_EQ(dy->data_type(), data_type);
    CHECK_EQ(dx->shape_view(), x->shape_view());
    CHECK_EQ(dx->data_type(), data_type);
    CHECK_GE(axis, 0);
    CHECK_LT(axis, x->shape_view().NumAxes());

    const CudnnTensorDescHelper desc_helper(x->shape_view(), data_type, axis,
                                            CUDNN_BATCHNORM_SPATIAL_PERSISTENT);
    desc_helper.CheckParamTensor(gamma);
    desc_helper.CheckParamTensor(beta);
    desc_helper.CheckParamTensor(gamma_diff);
    desc_helper.CheckParamTensor(beta_diff);
    desc_helper.CheckParamTensor(mean);
    desc_helper.CheckParamTensor(inv_variance);

    CudnnActivationDesc activation_desc(CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0);
    cudnnTensorDescriptor_t dz_desc;
    void* dz_ptr;
    cudnnBatchNormOps_t ops;
    if (ctx->has_output("addend_diff", 0)) {
      dz_desc = desc_helper.xy_desc();
      dz_ptr = ctx->Tensor4ArgNameAndIndex("addend_diff", 0)->mut_dptr();
      ops = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
    } else {
      dz_desc = nullptr;
      dz_ptr = nullptr;
      ops = CUDNN_BATCHNORM_OPS_BN_ACTIVATION;
    }

    size_t min_workspace_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationBackwardExWorkspaceSize(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
        ops, desc_helper.xy_desc(), desc_helper.xy_desc(), desc_helper.xy_desc(), dz_desc,
        desc_helper.xy_desc(), desc_helper.param_desc(), activation_desc.Get(),
        &min_workspace_size));
    const size_t workspace_size = tmp_buffer->shape_view().elem_cnt();
    CHECK_GE(workspace_size, min_workspace_size);
    size_t min_reserve_space_size;
    OF_CUDNN_CHECK(cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
        ops, activation_desc.Get(), desc_helper.xy_desc(), &min_reserve_space_size));
    const size_t reserve_space_size = reserve_space->shape_view().elem_cnt();
    CHECK_GE(reserve_space_size, min_reserve_space_size);
    OF_CUDNN_CHECK(cudnnBatchNormalizationBackwardEx(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), CUDNN_BATCHNORM_SPATIAL_PERSISTENT,
        ops, CudnnSPOnePtr(data_type), CudnnSPZeroPtr(data_type), CudnnSPOnePtr(data_type),
        CudnnSPZeroPtr(data_type), desc_helper.xy_desc(), x->dptr(), desc_helper.xy_desc(),
        y->dptr(), desc_helper.xy_desc(), dy->dptr(), dz_desc, dz_ptr, desc_helper.xy_desc(),
        dx->mut_dptr(), desc_helper.param_desc(), gamma->dptr(), beta->dptr(),
        gamma_diff->mut_dptr(), beta_diff->mut_dptr(), epsilon, mean->dptr(), inv_variance->dptr(),
        activation_desc.Get(), tmp_buffer->mut_dptr(), workspace_size,
        const_cast<void*>(reserve_space->dptr()), reserve_space_size));
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("cudnn_fused_normalization_add_relu_grad")
    .SetCreateFn<FusedNormalizationAddReluGradUserKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA))
    .SetInferTmpSizeFn(InferFusedNormalizationAddReluGradTmpSize);

#endif

}  // namespace
}  // namespace oneflow

#endif
