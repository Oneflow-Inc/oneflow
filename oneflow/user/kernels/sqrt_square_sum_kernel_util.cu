#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/sqrt_square_sum_kernel_util.h"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include <hipcub/hipcub.hpp>

namespace oneflow {

namespace {

template<typename T>
__global__ void SqrtSquareSumForOneThreadBlock(int64_t n, const T* x, T* y) {
  T t_sum = 0;
  CUDA_1D_KERNEL_LOOP(i, n) { t_sum += x[i] * x[i]; }
  typedef hipcub::BlockReduce<T, kCudaThreadsNumPerBlock> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  T b_sum = BlockReduce(temp_storage).Sum(t_sum);
  if (threadIdx.x == 0) { *y = sqrt(b_sum); }
}

template<typename T>
__global__ void SqrtSumForMultiThreadBlock(int64_t n, const T* x, T* y) {
  T t_sum = 0;
  CUDA_1D_KERNEL_LOOP(i, n) { t_sum += x[i]; }
  typedef hipcub::BlockReduce<T, kCudaThreadsNumPerBlock> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  T b_sum = BlockReduce(temp_storage).Sum(t_sum);
  if (threadIdx.x == 0) { *y = sqrt(b_sum); }
}

template<typename T>
__global__ void SquareSumForMultiThreadBlock(int64_t n, const T* x, T* tmp) {
  T t_sum = 0;
  CUDA_1D_KERNEL_LOOP(i, n) { t_sum += x[i] * x[i]; }
  typedef hipcub::BlockReduce<T, kCudaThreadsNumPerBlock> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  T b_sum = BlockReduce(temp_storage).Sum(t_sum);
  if (threadIdx.x == 0) { tmp[blockIdx.x] = b_sum; }
}

}  // namespace

template<typename T>
struct SqrtSquareSumKernelUtil<DeviceType::kCUDA, T> {
  static void SqrtSquareSum(ep::Stream* stream, int64_t n, const T* x, T* y, T* tmp) {
    const int32_t num_blocks = BlocksNum4ThreadsNum(n);
    CHECK_GE(num_blocks, 0);
    if (num_blocks == 1) {
      SqrtSquareSumForOneThreadBlock<T>
          <<<1, kCudaThreadsNumPerBlock, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(n, x, y);
    } else {
      Memset<DeviceType::kCUDA>(stream, y, 0, sizeof(T));
      SquareSumForMultiThreadBlock<T>
          <<<num_blocks, kCudaThreadsNumPerBlock, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
              n, x, tmp);
      SqrtSumForMultiThreadBlock<T>
          <<<1, kCudaThreadsNumPerBlock, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
              num_blocks, tmp, y);
    }
  }
};

#define INSTANTIATE_SQRT_SQUARE_SUM_KERNEL_UTIL_CUDA(type_cpp, type_proto) \
  template struct SqrtSquareSumKernelUtil<DeviceType::kCUDA, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_SQRT_SQUARE_SUM_KERNEL_UTIL_CUDA, FLOATING_DATA_TYPE_SEQ);
#undef INSTANTIATE_SQRT_SQUARE_SUM_KERNEL_UTIL_CUDA

}  // namespace oneflow
