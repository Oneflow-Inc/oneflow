#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/util/cuda_half_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void TriuGpu(const int64_t elem_cnt, const int64_t num_rows, const int64_t num_cols,
                        const int64_t diagonal, const T* x, T* y) {
  const int64_t matrix_size = num_rows * num_cols;
  CUDA_1D_KERNEL_LOOP_T(int64_t, k, elem_cnt) {
    const int64_t offset_in_matrix = k % matrix_size;
    const int64_t i = offset_in_matrix / num_cols;
    const int64_t j = offset_in_matrix - num_cols * i;
    y[k] = j < i + diagonal ? static_cast<T>(0) : x[k];
  }
}

template<typename T>
__global__ void TriuWarpProcessRowGpu(const int64_t total_rows, const int64_t num_rows,
                                      const int64_t num_cols, const int64_t diagonal, const T* x,
                                      T* y) {
  const int64_t warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / kCudaWarpSize;
  const int64_t lan_id = threadIdx.x % kCudaWarpSize;
  const int64_t num_warp = blockDim.x * gridDim.x / kCudaWarpSize;
  for (int64_t i = warp_id; i < total_rows; i += num_warp) {
    const int64_t row = i % num_rows;
    for (int64_t col = lan_id; col < num_cols; col += kCudaWarpSize) {
      const int64_t idx = i * num_cols + col;
      y[idx] = col < row + diagonal ? static_cast<T>(0) : x[idx];
    }
  }
}

template<>
__global__ void TriuWarpProcessRowGpu<half>(const int64_t total_rows, const int64_t num_rows,
                                            const int64_t num_cols, const int64_t diagonal,
                                            const half* x, half* y) {
  const int64_t h2_num_cols = num_cols / 2;
  const auto* x_h2 = reinterpret_cast<const half2*>(x);
  auto* y_h2 = reinterpret_cast<half2*>(y);

  const int64_t warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / kCudaWarpSize;
  const int64_t lan_id = threadIdx.x % kCudaWarpSize;
  const int64_t num_warp = blockDim.x * gridDim.x / kCudaWarpSize;
  for (int64_t i = warp_id; i < total_rows; i += num_warp) {
    const int64_t row = i % num_rows;
    for (int64_t col = lan_id; col < h2_num_cols; col += kCudaWarpSize) {
      const int64_t idx = i * h2_num_cols + col;
      const half2 x_val = x_h2[idx];
      half2 y_val;
      y_val.x = (2 * col) < row + diagonal ? static_cast<half>(0) : x_val.x;
      y_val.y = (2 * col + 1) < row + diagonal ? static_cast<half>(0) : x_val.y;
      y_h2[idx] = y_val;
    }
  }
}

}  // namespace

template<typename T>
class GpuTriuKernel final : public user_op::OpKernel {
 public:
  GpuTriuKernel() = default;
  ~GpuTriuKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("in", 0);
    const auto shape = x->shape_view();
    const auto diagonal = ctx->Attr<int64_t>("diagonal");
    const int64_t num_rows = shape.At(shape.NumAxes() - 2);
    const int64_t num_cols = shape.At(shape.NumAxes() - 1);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int32_t elem_cnt = shape.elem_cnt();
    if (elem_cnt == 0) { return; }
    if (num_cols % (kCudaWarpSize * 2) == 0) {
      const int64_t total_rows = elem_cnt / num_cols;
      TriuWarpProcessRowGpu<<<BlocksNum4ThreadsNum(total_rows * kCudaWarpSize),
                              kCudaThreadsNumPerBlock, 0,
                              ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          total_rows, num_rows, num_cols, diagonal, x->dptr<T>(), y->mut_dptr<T>());
    } else {
      TriuGpu<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          elem_cnt, num_rows, num_cols, diagonal, x->dptr<T>(), y->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_TRIU_KERNEL(dtype)                                                        \
  REGISTER_USER_KERNEL("triu")                                                                  \
      .SetCreateFn<GpuTriuKernel<dtype>>()                                                      \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                          \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value))        \
      .SetInplaceProposalFn([](const user_op::InferContext&,                                    \
                               user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> { \
        OF_RETURN_IF_ERROR(AddInplaceArgPairFn("out", 0, "in", 0, true));                       \
        return Maybe<void>::Ok();                                                               \
      });

REGISTER_CUDA_TRIU_KERNEL(half)
REGISTER_CUDA_TRIU_KERNEL(float)
REGISTER_CUDA_TRIU_KERNEL(double)
REGISTER_CUDA_TRIU_KERNEL(uint8_t)
REGISTER_CUDA_TRIU_KERNEL(int8_t)
REGISTER_CUDA_TRIU_KERNEL(int32_t)
REGISTER_CUDA_TRIU_KERNEL(int64_t)
REGISTER_CUDA_TRIU_KERNEL(bool)

}  // namespace oneflow
