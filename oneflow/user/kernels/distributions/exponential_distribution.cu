#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/common/data_type.h"
#include "oneflow/core/ep/include/device.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/user/kernels/distributions/exponential_distribution.h"
#include "oneflow/user/kernels/fused_rnn_cell_kernel_util.h"

namespace oneflow {

namespace {

// NOTE(Liang Depeng): the implementation of exponential cuda kernel is modified from
// https://github.com/pytorch/pytorch/blob/master/aten/src/ATen/native/cuda/DistributionTemplates.h

// launch bounds used for kernels
const uint32_t block_size_bound = 256;
const uint32_t grid_size_bound = 4;
// number of randoms given by distributions like hiprand_uniform4, hiprand_uniform2_double
// used in calculating philox offset.
const uint32_t curand4_engine_calls = 4;

std::tuple<uint64_t, dim3, dim3> calc_execution_policy(int64_t total_elements,
                                                       ep::CudaStream* stream) {
  const uint64_t numel = static_cast<uint64_t>(total_elements);
  const uint32_t block_size = block_size_bound;
  const uint32_t unroll = curand4_engine_calls;
  dim3 dim_block(block_size);
  dim3 grid((numel + block_size - 1) / block_size);
  uint32_t blocks_per_sm = stream->device_properties().maxThreadsPerMultiProcessor / block_size;
  grid.x = std::min(
      static_cast<uint32_t>(stream->device_properties().multiProcessorCount) * blocks_per_sm,
      grid.x);
  // number of times random will be generated per thread, to offset philox counter in thc random
  // state
  uint64_t counter_offset =
      ((numel - 1) / (block_size * grid.x * unroll) + 1) * curand4_engine_calls;
  return std::make_tuple(counter_offset, grid, dim_block);
}

OF_LAUNCH_BOUNDS_2(block_size_bound, grid_size_bound)
__global__ void distribution_elementwise_grid_stride_kernel_double(int32_t numel, uint64_t seed,
                                                                   uint64_t offset, double lambd,
                                                                   double epsilon,
                                                                   double* out_ptr) {
  int32_t unroll_factor = 2;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, offset, &state);

  int rounded_size = ((numel - 1) / (blockDim.x * gridDim.x * unroll_factor) + 1) * blockDim.x
                     * gridDim.x * unroll_factor;
  for (int32_t linear_index = idx; linear_index < rounded_size;
       linear_index += blockDim.x * gridDim.x * unroll_factor) {
    double2 rand = hiprand_uniform2_double(&state);
#pragma unroll
    for (int ii = 0; ii < unroll_factor; ii++) {
      int li = linear_index + blockDim.x * gridDim.x * ii;
      if (li < numel) {
        double log_rand = ::log(static_cast<double>((&rand.x)[ii]));
        // hiprand_uniform has (0,1] bounds. log(1) is 0 and exponential excludes 0.
        // we need log to be not 0, and not underflow when converted to half
        // fast __logf approximation can underflow, so set log to -epsilon/2 for 1 or close to 1
        // args
        double log = static_cast<double>((&rand.x)[ii]) >= static_cast<double>(1.) - epsilon / 2
                         ? -epsilon / 2
                         : log_rand;
        out_ptr[li] = static_cast<double>(-1.0) / lambd * log;
      }
    }
    __syncthreads();
  }
}

OF_LAUNCH_BOUNDS_2(block_size_bound, grid_size_bound)
__global__ void distribution_elementwise_grid_stride_kernel_float(int32_t numel, uint64_t seed,
                                                                  uint64_t offset, float lambd,
                                                                  float epsilon, float* out_ptr) {
  int32_t unroll_factor = 4;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, offset, &state);

  int rounded_size = ((numel - 1) / (blockDim.x * gridDim.x * unroll_factor) + 1) * blockDim.x
                     * gridDim.x * unroll_factor;
  for (int32_t linear_index = idx; linear_index < rounded_size;
       linear_index += blockDim.x * gridDim.x * unroll_factor) {
    float4 rand = hiprand_uniform4(&state);
#pragma unroll
    for (int ii = 0; ii < unroll_factor; ii++) {
      int li = linear_index + blockDim.x * gridDim.x * ii;
      if (li < numel) {
        float log_rand = __logf(static_cast<float>((&rand.x)[ii]));
        // hiprand_uniform has (0,1] bounds. log(1) is 0 and exponential excludes 0.
        // we need log to be not 0, and not underflow when converted to half
        // fast __logf approximation can underflow, so set log to -epsilon/2 for 1 or close to 1
        // args
        float log = static_cast<float>((&rand.x)[ii]) >= static_cast<float>(1.) - epsilon / 2
                        ? -epsilon / 2
                        : log_rand;
        out_ptr[li] = static_cast<float>(-1.0) / lambd * log;
      }
    }
    __syncthreads();
  }
}

}  // namespace

template<>
void ExponentialDistribution<DeviceType::kCUDA, double>::operator()(
    ep::Stream* stream, const int64_t elem_cnt, double* dptr,
    const std::shared_ptr<one::Generator>& generator) const {
  CHECK_GT(elem_cnt, 0);
  const auto device_index = stream->device()->device_index();
  auto gen = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>(device_index));
  ep::CudaStream* hip_stream = stream->As<ep::CudaStream>();
  auto execution_policy = calc_execution_policy(elem_cnt, hip_stream);

  auto counter_offset = std::get<0>(execution_policy);
  auto grid = std::get<1>(execution_policy);
  auto block = std::get<2>(execution_policy);

  uint64_t offset = 0;
  uint64_t seed = gen->current_seed();
  {
    std::lock_guard<std::mutex> lock(gen->mutex_);
    offset = gen->get_philox_offset(counter_offset);
  }

  distribution_elementwise_grid_stride_kernel_double<<<
      grid, block, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
      elem_cnt, seed, offset, lambd_, std::numeric_limits<double>::epsilon(), dptr);
}

template<>
void ExponentialDistribution<DeviceType::kCUDA, float>::operator()(
    ep::Stream* stream, const int64_t elem_cnt, float* dptr,
    const std::shared_ptr<one::Generator>& generator) const {
  CHECK_GT(elem_cnt, 0);
  const auto device_index = stream->device()->device_index();
  auto gen = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>(device_index));
  ep::CudaStream* hip_stream = stream->As<ep::CudaStream>();
  auto execution_policy = calc_execution_policy(elem_cnt, hip_stream);

  auto counter_offset = std::get<0>(execution_policy);
  auto grid = std::get<1>(execution_policy);
  auto block = std::get<2>(execution_policy);

  uint64_t offset = 0;
  uint64_t seed = gen->current_seed();
  {
    std::lock_guard<std::mutex> lock(gen->mutex_);
    offset = gen->get_philox_offset(counter_offset);
  }

  distribution_elementwise_grid_stride_kernel_float<<<
      grid, block, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
      elem_cnt, seed, offset, lambd_, std::numeric_limits<float>::epsilon(), dptr);
}

}  // namespace oneflow
