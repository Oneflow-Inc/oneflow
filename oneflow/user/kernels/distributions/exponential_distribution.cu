/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/common/data_type.h"
#include "oneflow/core/ep/include/device.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/user/kernels/distributions/distribution_template_util.cuh"
#include "oneflow/user/kernels/distributions/exponential_distribution.h"
#include "oneflow/user/kernels/fused_rnn_cell_kernel_util.h"

namespace oneflow {

template<>
void ExponentialDistribution<DeviceType::kCUDA, double>::operator()(
    ep::Stream* stream, const int64_t elem_cnt, double* dptr,
    const std::shared_ptr<one::Generator>& generator) const {
  CHECK_GT(elem_cnt, 0);
  const auto device_index = stream->device()->device_index();
  auto gen = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>(device_index));
  ep::CudaStream* hip_stream = stream->As<ep::CudaStream>();
  auto execution_policy = CalcExecutionPolicy(elem_cnt, hip_stream);

  auto counter_offset = std::get<0>(execution_policy);
  auto grid = std::get<1>(execution_policy);
  auto block = std::get<2>(execution_policy);

  uint64_t offset = 0;
  uint64_t seed = gen->current_seed();
  {
    std::lock_guard<std::mutex> lock(gen->mutex_);
    offset = gen->get_philox_offset(counter_offset);
  }

  DistributionElementwiseGridStrideParams params;
  params.numel = elem_cnt;
  params.seed = seed;
  params.offset = offset;
  params.dst = dptr;
  params.attr0 = Scalar(std::numeric_limits<double>::epsilon());
  params.attr1 = Scalar(lambd_);

  DistributionElementwiseGridStrideKernel<double, double, 2, DistributionOp::kUniform2Double,
                                          TransformOp::kExponential>
      <<<grid, block, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(params);
}

template<>
void ExponentialDistribution<DeviceType::kCUDA, float>::operator()(
    ep::Stream* stream, const int64_t elem_cnt, float* dptr,
    const std::shared_ptr<one::Generator>& generator) const {
  CHECK_GT(elem_cnt, 0);
  const auto device_index = stream->device()->device_index();
  auto gen = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>(device_index));
  ep::CudaStream* hip_stream = stream->As<ep::CudaStream>();
  auto execution_policy = CalcExecutionPolicy(elem_cnt, hip_stream);

  auto counter_offset = std::get<0>(execution_policy);
  auto grid = std::get<1>(execution_policy);
  auto block = std::get<2>(execution_policy);

  uint64_t offset = 0;
  uint64_t seed = gen->current_seed();
  {
    std::lock_guard<std::mutex> lock(gen->mutex_);
    offset = gen->get_philox_offset(counter_offset);
  }

  DistributionElementwiseGridStrideParams params;
  params.numel = elem_cnt;
  params.seed = seed;
  params.offset = offset;
  params.dst = dptr;
  params.attr0 = Scalar(std::numeric_limits<float>::epsilon());
  params.attr1 = Scalar(lambd_);

  DistributionElementwiseGridStrideKernel<float, float, 4, DistributionOp::kUniform4,
                                          TransformOp::kExponential>
      <<<grid, block, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(params);
}

}  // namespace oneflow
