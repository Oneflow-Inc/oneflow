#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/user/kernels/distributions/normal_distribution.h"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/ep/include/device.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/user/kernels/fused_rnn_cell_kernel_util.h"
#include "oneflow/core/cuda/layer_norm.cuh"

namespace oneflow {

namespace {

// launch bounds used for kernels
const uint32_t block_size_bound = 256;
const uint32_t grid_size_bound = 4;

std::tuple<uint64_t, dim3, dim3> CalcExecutionPolicy(int64_t total_elements,
                                                     ep::CudaStream* stream) {
  const uint64_t numel = static_cast<uint64_t>(total_elements);
  const uint32_t block_size = block_size_bound;
  // number of randoms given by distributions like hiprand_uniform4, hiprand_uniform2_double
  // used in calculating philox offset.
  const uint32_t curand4_engine_calls = 4;
  const uint32_t unroll = curand4_engine_calls;
  dim3 dim_block(block_size);
  dim3 grid((numel + block_size - 1) / block_size);
  uint32_t blocks_per_sm = stream->device_properties().maxThreadsPerMultiProcessor / block_size;
  grid.x = std::min(
      static_cast<uint32_t>(stream->device_properties().multiProcessorCount) * blocks_per_sm,
      grid.x);
  // number of times random will be generated per thread, to offset philox counter in thc random
  // state
  uint64_t counter_offset =
      ((numel - 1) / (block_size * grid.x * unroll) + 1) * curand4_engine_calls;
  return std::make_tuple(counter_offset, grid, dim_block);
}

template<typename T, typename ComputeType, int unroll_factor>
OF_LAUNCH_BOUNDS_2(block_size_bound, grid_size_bound)
__global__ void DistributionElementwiseGridStrideKernelDouble(int32_t numel, uint64_t seed,
                                                              uint64_t offset, ComputeType mean,
                                                              ComputeType std, T* out_ptr) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, offset, &state);

  int rounded_size = ((numel - 1) / (blockDim.x * gridDim.x * unroll_factor) + 1) * blockDim.x
                     * gridDim.x * unroll_factor;
  for (int32_t linear_index = idx; linear_index < rounded_size;
       linear_index += blockDim.x * gridDim.x * unroll_factor) {
    double2 rand = hiprand_normal2_double(&state);
#pragma unroll
    for (int ii = 0; ii < unroll_factor; ii++) {
      int li = linear_index + blockDim.x * gridDim.x * ii;
      if (li < numel) {
        out_ptr[li] = static_cast<T>(static_cast<ComputeType>((&rand.x)[ii]) * std + mean);
      }
    }
    __syncthreads();
  }
}

template<typename T, typename ComputeType, int unroll_factor>
OF_LAUNCH_BOUNDS_2(block_size_bound, grid_size_bound)
__global__ void DistributionElementwiseGridStrideKernelFloat(int32_t numel, uint64_t seed,
                                                             uint64_t offset, ComputeType mean,
                                                             ComputeType std, T* out_ptr) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, offset, &state);

  int rounded_size = ((numel - 1) / (blockDim.x * gridDim.x * unroll_factor) + 1) * blockDim.x
                     * gridDim.x * unroll_factor;
  for (int32_t linear_index = idx; linear_index < rounded_size;
       linear_index += blockDim.x * gridDim.x * unroll_factor) {
    float4 rand = hiprand_normal4(&state);
#pragma unroll
    for (int ii = 0; ii < unroll_factor; ii++) {
      int li = linear_index + blockDim.x * gridDim.x * ii;
      if (li < numel) {
        out_ptr[li] = static_cast<T>(static_cast<ComputeType>((&rand.x)[ii]) * std + mean);
      }
    }
    __syncthreads();
  }
}

}  // namespace

template<typename T>
void NormalDistribution<DeviceType::kCUDA, T>::operator()(
    ep::Stream* stream, const int64_t elem_cnt, T* dptr,
    const std::shared_ptr<one::Generator>& generator) const {
  CHECK_GE(elem_cnt, 0);
  if (elem_cnt == 0) return;
  const auto device_index = stream->device()->device_index();
  auto gen = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>(device_index));

  ep::CudaStream* hip_stream = stream->As<ep::CudaStream>();
  auto execution_policy = CalcExecutionPolicy(elem_cnt, hip_stream);

  auto counter_offset = std::get<0>(execution_policy);
  auto grid = std::get<1>(execution_policy);
  auto block = std::get<2>(execution_policy);

  uint64_t offset = 0;
  uint64_t seed = gen->current_seed();
  {
    std::lock_guard<std::mutex> lock(gen->mutex_);
    offset = gen->get_philox_offset(counter_offset);
  }

  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  if (std::is_same<T, double>::value) {
    DistributionElementwiseGridStrideKernelDouble<T, ComputeType, 2>
        <<<grid, block, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            elem_cnt, seed, offset, static_cast<ComputeType>(mean_), static_cast<ComputeType>(std_),
            dptr);
  } else {
    DistributionElementwiseGridStrideKernelFloat<T, ComputeType, 4>
        <<<grid, block, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            elem_cnt, seed, offset, static_cast<ComputeType>(mean_), static_cast<ComputeType>(std_),
            dptr);
  }
}

#define INITIATE_CUDA_NORMAL_DISTRIBUTION(T, typeproto)               \
  template void NormalDistribution<DeviceType::kCUDA, T>::operator()( \
      ep::Stream* stream, const int64_t elem_cnt, T* dptr,            \
      const std::shared_ptr<one::Generator>& generator) const;

OF_PP_FOR_EACH_TUPLE(INITIATE_CUDA_NORMAL_DISTRIBUTION, FLOATING_DATA_TYPE_SEQ)
INITIATE_CUDA_NORMAL_DISTRIBUTION(half, DataType::kFloat16)

}  // namespace oneflow
