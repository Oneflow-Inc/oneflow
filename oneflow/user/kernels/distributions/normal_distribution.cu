/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/user/kernels/distributions/distribution_template_util.cuh"
#include "oneflow/user/kernels/distributions/normal_distribution.h"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/ep/include/device.h"

namespace oneflow {

template<typename T>
void NormalDistribution<DeviceType::kCUDA, T>::operator()(
    ep::Stream* stream, const int64_t elem_cnt, T* dptr,
    const std::shared_ptr<one::Generator>& generator) const {
  CHECK_GE(elem_cnt, 0);
  if (elem_cnt == 0) return;
  const auto device_index = stream->device()->device_index();
  auto gen = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>(device_index));

  ep::CudaStream* hip_stream = stream->As<ep::CudaStream>();
  auto execution_policy = CalcExecutionPolicy(elem_cnt, hip_stream);

  auto counter_offset = std::get<0>(execution_policy);
  auto grid = std::get<1>(execution_policy);
  auto block = std::get<2>(execution_policy);

  uint64_t offset = 0;
  uint64_t seed = gen->current_seed();
  {
    std::lock_guard<std::mutex> lock(gen->mutex_);
    offset = gen->get_philox_offset(counter_offset);
  }

  using ComputeType = typename distribution::DefaultComputeType<T>::type;
  ComputeType mean = static_cast<ComputeType>(mean_);
  ComputeType std = static_cast<ComputeType>(std_);
  auto transform_func = [mean, std] __device__(ComputeType random_val) -> T {
    return static_cast<T>(random_val * std + mean);
  };
  if (std::is_same<T, double>::value) {
    DistributionElementwiseGridStrideKernel<T, 2>
        <<<grid, block, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            elem_cnt, seed, offset, dptr,
            [] __device__(hiprandStatePhilox4_32_10_t * state) {
              return hiprand_normal2_double(state);
            },
            transform_func);
  } else {
    DistributionElementwiseGridStrideKernel<T, 4>
        <<<grid, block, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            elem_cnt, seed, offset, dptr,
            [] __device__(hiprandStatePhilox4_32_10_t * state) { return hiprand_normal4(state); },
            transform_func);
  }
}

#define INITIATE_CUDA_NORMAL_DISTRIBUTION(T, typeproto)               \
  template void NormalDistribution<DeviceType::kCUDA, T>::operator()( \
      ep::Stream* stream, const int64_t elem_cnt, T* dptr,            \
      const std::shared_ptr<one::Generator>& generator) const;

OF_PP_FOR_EACH_TUPLE(INITIATE_CUDA_NORMAL_DISTRIBUTION, FLOATING_DATA_TYPE_SEQ)
INITIATE_CUDA_NORMAL_DISTRIBUTION(half, DataType::kFloat16)

}  // namespace oneflow
