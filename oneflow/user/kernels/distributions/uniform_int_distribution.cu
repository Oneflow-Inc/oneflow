/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/common/preprocessor.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/framework/dtype.h"
#include "oneflow/user/kernels/distributions/uniform_int_distribution.h"
#include "oneflow/user/kernels/distributions/distribution_template_util.cuh"
#include "oneflow/core/ep/include/device.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

template<typename T>
void UniformIntDistribution<DeviceType::kCUDA, T>::operator()(
    ep::Stream* stream, const int64_t elem_cnt, T* dptr,
    const std::shared_ptr<one::Generator>& generator) const {
  CHECK_GE(elem_cnt, 0);
  if (elem_cnt == 0) return;
  const auto device_index = stream->device()->device_index();
  auto gen = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>(device_index));

  ep::CudaStream* hip_stream = stream->As<ep::CudaStream>();
  auto execution_policy = CalcExecutionPolicy(elem_cnt, hip_stream);

  auto counter_offset = std::get<0>(execution_policy);
  auto grid = std::get<1>(execution_policy);
  auto block = std::get<2>(execution_policy);

  uint64_t offset = 0;
  uint64_t seed = gen->current_seed();
  {
    std::lock_guard<std::mutex> lock(gen->mutex_);
    offset = gen->get_philox_offset(counter_offset);
  }

  DistributionElementwiseGridStrideParams params;
  params.numel = elem_cnt;
  params.seed = seed;
  params.offset = offset;
  params.dst = reinterpret_cast<void*>(dptr);
  params.attr0 = Scalar(low_);
  params.attr1 = Scalar(high_);

  using ComputeType = typename distribution::DefaultComputeType<T>::type;

  if (std::is_same<T, double>::value) {
    DistributionElementwiseGridStrideKernel<T, ComputeType, 2, DistributionOp::kUniform2Double,
                                            TransformOp::kUniformInt>
        <<<grid, block, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(params);
  } else {
    DistributionElementwiseGridStrideKernel<T, ComputeType, 4, DistributionOp::kUniform4,
                                            TransformOp::kUniformInt>
        <<<grid, block, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(params);
  }
}

#define INITIATE_CUDA_UNIFORM_INT_DISTRIBUTION(T, typeproto)              \
  template void UniformIntDistribution<DeviceType::kCUDA, T>::operator()( \
      ep::Stream* stream, const int64_t elem_cnt, T* dptr,                \
      const std::shared_ptr<one::Generator>& generator) const;

OF_PP_FOR_EACH_TUPLE(INITIATE_CUDA_UNIFORM_INT_DISTRIBUTION, FLOATING_DATA_TYPE_SEQ)
OF_PP_FOR_EACH_TUPLE(INITIATE_CUDA_UNIFORM_INT_DISTRIBUTION, INT_DATA_TYPE_SEQ)
OF_PP_FOR_EACH_TUPLE(INITIATE_CUDA_UNIFORM_INT_DISTRIBUTION, UNSIGNED_INT_DATA_TYPE_SEQ)

}  // namespace oneflow
