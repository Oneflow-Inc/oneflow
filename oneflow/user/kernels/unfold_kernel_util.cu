
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#ifdef WITH_CUDA

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/user/kernels/unfold_kernel_util.h"
#include "oneflow/core/ep/cuda/cuda_stream.h"

namespace oneflow {

namespace user_op {

namespace {

constexpr int kBlockSize = cuda::elementwise::kBlockSize;

int GetNumBlocks(int64_t elem_cnt) {
  int num_blocks = 0;
  OF_CUDA_CHECK(cuda::elementwise::GetNumBlocks(elem_cnt, &num_blocks));
  return num_blocks;
}

// NDIM range: (1, 2, 3)
// SDIM range: (1, 2), 1 indicates channels_last, 2 indicates channels_first
template<typename T, typename INDEX_T, int NDIM, int SDIM>
__global__ void CudaUnfoldForward(UnfoldParams<INDEX_T, NDIM, SDIM> params, const T* in, T* out) {
  CUDA_1D_KERNEL_LOOP_T(INDEX_T, out_offset, params.out_elem_cnt) {
    using ParamType = UnfoldParams<INDEX_T, NDIM, SDIM>;
    INDEX_T in_index[ParamType::kInputNDim] = {0};
    INDEX_T out_index[ParamType::kOutputNDim] = {0};
    params.out_index_helper.OffsetToNdIndex(out_offset, out_index);
    if (!UnfoldIndexTransform<INDEX_T, NDIM, SDIM>(params, out_index, in_index)) {
      INDEX_T in_offset = params.in_index_helper.NdIndexToOffset(in_index);
      out[out_offset] = in[in_offset];
    } else {
      out[out_offset] = static_cast<T>(kUnfoldPaddingValue);
    }
  }
}

}  // namespace

template<typename T, typename INDEX_T, int NDIM, int SDIM>
struct UnfoldKernelUtil<DeviceType::kCUDA, T, INDEX_T, NDIM, SDIM> {
  using ParamType = UnfoldParams<INDEX_T, NDIM, SDIM>;
  static void Forward(ep::Stream* stream, const UnfoldParams<INDEX_T, NDIM, SDIM>* params,
                      const T* input_ptr, T* output_ptr) {
    CudaUnfoldForward<T, INDEX_T, NDIM, SDIM>
        <<<GetNumBlocks(params->out_elem_cnt), kBlockSize, 0,
           stream->As<ep::CudaStream>()->cuda_stream()>>>(*params, input_ptr, output_ptr);
  }
};
INSTANTIATE_UNFOLD_KERNEL_UTIL_FOR_DEVICE(DeviceType::kCUDA)
}  // namespace user_op
}  // namespace oneflow
#endif  // WITH_CUDA