#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

template<typename T, typename U>
__global__ void FusedCastScaleGpu(const int64_t n, const T scale_val, const U* in,
                                  const T* scale_by_ptr, T* out) {
  const T scale = *scale_by_ptr * scale_val;
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = static_cast<T>(in[i]) * scale; }
}

template<>
__global__ void FusedCastScaleGpu<float, half>(const int64_t n, const float scale_val,
                                               const half* in, const float* scale_by_ptr,
                                               float* out) {
  const float scale = *scale_by_ptr * scale_val;
  const int64_t n_2 = n / 2;
  const auto* in_2 = reinterpret_cast<const half2*>(in);
  auto* out_2 = reinterpret_cast<float2*>(out);
  CUDA_1D_KERNEL_LOOP(i, n_2) {
    float2 f2 = __half22float2(in_2[i]);
    f2.x *= scale;
    f2.y *= scale;
    out_2[i] = f2;
  }
  if (n % 2 == 1 && blockIdx.x == 0 && threadIdx.x == 0) {
    out[n - 1] = __half2float(in[n - 1]) * scale;
  }
}

template<>
__global__ void FusedCastScaleGpu<half, float>(const int64_t n, const half scale_val,
                                               const float* in, const half* scale_by_ptr,
                                               half* out) {
  const half scale = *scale_by_ptr * scale_val;
  const half2 scale_h2 = __half2half2(scale);
  const int64_t n_2 = n / 2;
  const auto* in_2 = reinterpret_cast<const float2*>(in);
  auto* out_h2 = reinterpret_cast<half2*>(out);
  CUDA_1D_KERNEL_LOOP(i, n_2) {
    half2 in_h2 = __float22half2_rn(in_2[i]);
    out_h2[i] = __hmul2(in_h2, scale_h2);
  }
  if (n % 2 == 1 && blockIdx.x == 0 && threadIdx.x == 0) {
    out[n - 1] = __float2half(in[n - 1]) * scale;
  }
}

template<typename T, typename U>
class FusedCastScaleGpuKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  FusedCastScaleGpuKernel() = default;
  ~FusedCastScaleGpuKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const int64_t n = x->shape_view().elem_cnt();
    const double scale = ctx->Attr<double>("scale");
    const int64_t launch_n = ((std::is_same<T, half>::value && std::is_same<U, float>::value)
                              || (std::is_same<T, float>::value && std::is_same<U, half>::value))
                                 ? RoundUp(n, 2) / 2
                                 : n;
    FusedCastScaleGpu<T, U><<<BlocksNum4ThreadsNum(launch_n), kCudaThreadsNumPerBlock, 0,
                              ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        n, static_cast<T>(scale), x->dptr<U>(), scale_by_tensor->dptr<T>(), y->mut_dptr<T>());
  };
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

}  // namespace

#define REGISTER_FUSED_CAST_SCALE_CUDA_KERNEL(x_type, y_type)                          \
  REGISTER_USER_KERNEL("fused_cast_scale")                                             \
      .SetCreateFn<FusedCastScaleGpuKernel<y_type, x_type>>()                          \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                 \
                       && (user_op::HobDataType("y", 0) == GetDataType<y_type>::value) \
                       && (user_op::HobDataType("x", 0) == GetDataType<x_type>::value));

REGISTER_FUSED_CAST_SCALE_CUDA_KERNEL(half, float);
REGISTER_FUSED_CAST_SCALE_CUDA_KERNEL(half, double);
REGISTER_FUSED_CAST_SCALE_CUDA_KERNEL(float, half);
REGISTER_FUSED_CAST_SCALE_CUDA_KERNEL(float, double);
REGISTER_FUSED_CAST_SCALE_CUDA_KERNEL(double, half);
REGISTER_FUSED_CAST_SCALE_CUDA_KERNEL(double, float);
#undef REGISTER_FUSED_CAST_SCALE_CUDA_KERNEL

}  // namespace oneflow
