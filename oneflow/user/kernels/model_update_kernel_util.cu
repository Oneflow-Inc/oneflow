#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/user/kernels/model_update_kernel_util.h"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

template<typename T, typename G, typename C>
__global__ void SGDUpdateGpu(int64_t n, T scale, float l1, float l2, float weight_decay,
                             float learning_rate_val, float lr_scale, const float* learning_rate,
                             const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff,
                             T* model, C* model_copy) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
  if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
  learning_rate_val *= lr_scale;
  CUDA_1D_KERNEL_LOOP(i, n) {
    if (model_copy != nullptr) {
      FusedSGDUpdateFunctor<T, G, C>()(model_diff + i, model + i, model_copy + i, scale, l1, l2,
                                       weight_decay, learning_rate_val);
    } else {
      SGDUpdateFunctor<T, G>()(model_diff + i, model + i, scale, l1, l2, weight_decay,
                               learning_rate_val);
    }
  }
}

template<typename T, typename K, typename IDX>
__global__ void IndexedSlicesSGDUpdateGpu(float weight_decay, float lr_scale,
                                          const IDX feature_size, const int64_t lower_bound,
                                          const int64_t upper_bound, const IDX* num_unique_instance,
                                          const float* learning_rate, const K* indices,
                                          const T* values, T* model) {
  const int64_t n = *num_unique_instance * feature_size;
  T lr = *learning_rate;
  lr *= lr_scale;
  CUDA_1D_KERNEL_LOOP_T(IDX, i, n) {
    const IDX indices_idx = i / feature_size;
    const IDX inner_idx = i - indices_idx * feature_size;
    const IDX instance_id = indices[indices_idx];
    if (instance_id >= lower_bound && instance_id < upper_bound) {
      const IDX model_idx = (instance_id - lower_bound) * feature_size + inner_idx;
      SGDUpdateFunctor<T, T>()(values + i, model + model_idx, static_cast<T>(1), 0.0, 0.0,
                               weight_decay, lr);
    }
  }
}

template<typename T>
__global__ void SumSquares2(int64_t n, const T* src0, T* dst0, const T* src1, T* dst1) {
  T t_sum0 = 0;
  T t_sum1 = 0;
  CUDA_1D_KERNEL_LOOP(i, n) {
    t_sum0 += src0[i] * src0[i];
    t_sum1 += src1[i] * src1[i];
  }
  typedef hipcub::BlockReduce<T, kCudaThreadsNumPerBlock> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage0;
  __shared__ typename BlockReduce::TempStorage temp_storage1;
  T b_sum0 = BlockReduce(temp_storage0).Sum(t_sum0);
  T b_sum1 = BlockReduce(temp_storage1).Sum(t_sum1);
  if (threadIdx.x == 0) {
    cuda::atomic::Add(dst0, b_sum0);
    cuda::atomic::Add(dst1, b_sum1);
  }
}

}  // namespace

template<typename T, typename G, typename C>
struct SGDUpdateKernelUtil<DeviceType::kCUDA, T, G, C> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float weight_decay,
                     float learning_rate_val, float lr_scale, const float* learning_rate,
                     const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff, T* model,
                     C* model_copy);
};

template<typename T, typename G, typename C>
void SGDUpdateKernelUtil<DeviceType::kCUDA, T, G, C>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float weight_decay,
    float learning_rate_val, float lr_scale, const float* learning_rate, const T* scale_by_ptr,
    const int64_t* skip_if, const G* model_diff, T* model, C* model_copy) {
  SGDUpdateGpu<T, G, C><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                          stream->As<ep::CudaStream>()->hip_stream()>>>(
      n, scale, l1, l2, weight_decay, learning_rate_val, lr_scale, learning_rate, scale_by_ptr,
      skip_if, model_diff, model, model_copy);
}

template<typename T, typename G>
struct SGDUpdateKernelUtil<DeviceType::kCUDA, T, G, float16> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float weight_decay,
                     float learning_rate_val, float lr_scale, const float* learning_rate,
                     const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff, T* model,
                     float16* model_copy);
};

template<typename T, typename G>
void SGDUpdateKernelUtil<DeviceType::kCUDA, T, G, float16>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float weight_decay,
    float learning_rate_val, float lr_scale, const float* learning_rate, const T* scale_by_ptr,
    const int64_t* skip_if, const G* model_diff, T* model, float16* model_copy) {
  SGDUpdateKernelUtil<DeviceType::kCUDA, T, G, half>::Update(
      stream, n, scale, l1, l2, weight_decay, learning_rate_val, lr_scale, learning_rate,
      scale_by_ptr, skip_if, model_diff, model, reinterpret_cast<half*>(model_copy));
}

template<typename T>
struct SGDUpdateKernelUtil<DeviceType::kCUDA, T, float16, float16> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float weight_decay,
                     float learning_rate_val, float lr_scale, const float* learning_rate,
                     const T* scale_by_ptr, const int64_t* skip_if, const float16* model_diff,
                     T* model, float16* model_copy);
};

template<typename T>
void SGDUpdateKernelUtil<DeviceType::kCUDA, T, float16, float16>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float weight_decay,
    float learning_rate_val, float lr_scale, const float* learning_rate, const T* scale_by_ptr,
    const int64_t* skip_if, const float16* model_diff, T* model, float16* model_copy) {
  SGDUpdateKernelUtil<DeviceType::kCUDA, T, half, half>::Update(
      stream, n, scale, l1, l2, weight_decay, learning_rate_val, lr_scale, learning_rate,
      scale_by_ptr, skip_if, reinterpret_cast<const half*>(model_diff), model,
      reinterpret_cast<half*>(model_copy));
}

template struct SGDUpdateKernelUtil<DeviceType::kCUDA, double, double, float16>;
template struct SGDUpdateKernelUtil<DeviceType::kCUDA, float, float, float16>;
template struct SGDUpdateKernelUtil<DeviceType::kCUDA, float, float16, float16>;

template<typename T, typename K, typename IDX>
struct IndexedSlicesSGDUpdateKernelUtil<DeviceType::kCUDA, T, K, IDX> {
  static void Update(ep::Stream* stream, float weight_decay, float lr_scale, int64_t num_indices,
                     int64_t feature_size, int64_t lower_bound, int64_t upper_bound,
                     const IDX* num_unique_instance, const float* learning_rate, const K* indices,
                     const T* values, T* model);
};

template<typename T, typename K, typename IDX>
void IndexedSlicesSGDUpdateKernelUtil<DeviceType::kCUDA, T, K, IDX>::Update(
    ep::Stream* stream, float weight_decay, float lr_scale, int64_t num_indices,
    int64_t feature_size, int64_t lower_bound, int64_t upper_bound, const IDX* num_unique_instance,
    const float* learning_rate, const K* indices, const T* values, T* model) {
  IndexedSlicesSGDUpdateGpu<T, K, IDX>
      <<<BlocksNum4ThreadsNum(num_indices * feature_size), kCudaThreadsNumPerBlock, 0,
         stream->As<ep::CudaStream>()->hip_stream()>>>(
          weight_decay, lr_scale, feature_size, lower_bound, upper_bound, num_unique_instance,
          learning_rate, indices, values, model);
}

#define INITIATE_INDEXED_SLICES_SGD_UPDATE_KERNEL_UTIL_CUDA(val_type_pair, key_type_pair,  \
                                                            idx_type_pair)                 \
  template struct IndexedSlicesSGDUpdateKernelUtil<                                        \
      DeviceType::kCUDA, OF_PP_PAIR_FIRST(val_type_pair), OF_PP_PAIR_FIRST(key_type_pair), \
      OF_PP_PAIR_FIRST(idx_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INITIATE_INDEXED_SLICES_SGD_UPDATE_KERNEL_UTIL_CUDA,
                                 FLOATING_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ, INT_DATA_TYPE_SEQ);
#undef INITIATE_INDEXED_SLICES_SGD_UPDATE_KERNEL_UTIL_CUDA

namespace {

template<typename T, typename G>
__global__ void MomentumUpdateGpu(int64_t n, T scale, float l1, float l2, float beta,
                                  float dampening, bool nesterov, bool maximize, float weight_decay,
                                  float learning_rate_val, float lr_scale,
                                  const float* learning_rate, const T* scale_by_ptr,
                                  const int64_t* skip_if, const G* model_diff, T* model,
                                  T* momentum) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
  if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
  learning_rate_val *= lr_scale;
  CUDA_1D_KERNEL_LOOP(i, n) {
    MomentumUpdateFunctor<T, G>()(model_diff + i, model + i, momentum + i, scale, l1, l2, beta,
                                  dampening, nesterov, maximize, weight_decay, learning_rate_val);
  }
}

template<typename T, typename K, typename IDX>
__global__ void IndexedSlicesMomentumUpdateGpu(T beta, float dampening, bool nesterov,
                                               bool maximize, float weight_decay, float lr_scale,
                                               int64_t feature_size, int64_t lower_bound,
                                               int64_t upper_bound, const IDX* num_unique_instance,
                                               const float* learning_rate, const K* indices,
                                               const T* values, T* model, T* momentum) {
  const int64_t n = *num_unique_instance * feature_size;
  T lr = *learning_rate;
  lr *= lr_scale;
  CUDA_1D_KERNEL_LOOP(i, n) {
    const IDX indices_idx = i / feature_size;
    const IDX inner_idx = i - indices_idx * feature_size;
    const IDX instance_id = indices[indices_idx];
    if (instance_id >= lower_bound && instance_id < upper_bound) {
      const IDX model_idx = (instance_id - lower_bound) * feature_size + inner_idx;
      MomentumUpdateFunctor<T, T>()(values + i, model + model_idx, momentum + model_idx,
                                    static_cast<T>(1), 0.0, 0.0, beta, dampening, nesterov,
                                    maximize, weight_decay, lr);
    }
  }
}
}  // namespace

template<typename T, typename G>
struct MomentumUpdateKernelUtil<DeviceType::kCUDA, T, G> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float beta,
                     float dampening, bool nesterov, bool maximize, float weight_decay,
                     float learning_rate_val, float lr_scale, const float* learning_rate,
                     const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff, T* model,
                     T* momentum);
};

template<typename T, typename G>
void MomentumUpdateKernelUtil<DeviceType::kCUDA, T, G>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float beta, float dampening,
    bool nesterov, bool maximize, float weight_decay, float learning_rate_val, float lr_scale,
    const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff,
    T* model, T* momentum) {
  MomentumUpdateGpu<T, G><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                            stream->As<ep::CudaStream>()->hip_stream()>>>(
      n, scale, l1, l2, beta, dampening, nesterov, maximize, weight_decay, learning_rate_val,
      lr_scale, learning_rate, scale_by_ptr, skip_if, model_diff, model, momentum);
}

template<typename T>
struct MomentumUpdateKernelUtil<DeviceType::kCUDA, T, float16> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float beta,
                     float dampening, bool nesterov, bool maximize, float weight_decay,
                     float learning_rate_val, float lr_scale, const float* learning_rate,
                     const T* scale_by_ptr, const int64_t* skip_if, const float16* model_diff,
                     T* model, T* momentum);
};

template<typename T>
void MomentumUpdateKernelUtil<DeviceType::kCUDA, T, float16>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float beta, float dampening,
    bool nesterov, bool maximize, float weight_decay, float learning_rate_val, float lr_scale,
    const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if,
    const float16* model_diff, T* model, T* momentum) {
  MomentumUpdateKernelUtil<DeviceType::kCUDA, T, half>::Update(
      stream, n, scale, l1, l2, beta, dampening, nesterov, maximize, weight_decay,
      learning_rate_val, lr_scale, learning_rate, scale_by_ptr, skip_if,
      reinterpret_cast<const half*>(model_diff), model, momentum);
}

template struct MomentumUpdateKernelUtil<DeviceType::kCUDA, double, double>;
template struct MomentumUpdateKernelUtil<DeviceType::kCUDA, float, float>;
template struct MomentumUpdateKernelUtil<DeviceType::kCUDA, float, float16>;

template<typename T, typename K, typename IDX>
struct IndexedSlicesMomentumMdUpdateKernelUtil<DeviceType::kCUDA, T, K, IDX> {
  static void Update(ep::Stream* stream, T beta, float dampening, bool nesterov, bool maximize,
                     float weight_decay, float lr_scale, int64_t num_instance, int64_t feature_size,
                     int64_t lower_bound, int64_t upper_bound, const IDX* num_unique_instance,
                     const float* learning_rate, const K* indices, const T* values, T* model,
                     T* momentum);
};

template<typename T, typename K, typename IDX>
void IndexedSlicesMomentumMdUpdateKernelUtil<DeviceType::kCUDA, T, K, IDX>::Update(
    ep::Stream* stream, T beta, float dampening, bool nesterov, bool maximize, float weight_decay,
    float lr_scale, int64_t num_instance, int64_t feature_size, int64_t lower_bound,
    int64_t upper_bound, const IDX* num_unique_instance, const float* learning_rate,
    const K* indices, const T* values, T* model, T* momentum) {
  IndexedSlicesMomentumUpdateGpu<T, K, IDX>
      <<<BlocksNum4ThreadsNum(num_instance * feature_size), kCudaThreadsNumPerBlock, 0,
         stream->As<ep::CudaStream>()->hip_stream()>>>(
          beta, dampening, nesterov, maximize, weight_decay, lr_scale, feature_size, lower_bound,
          upper_bound, num_unique_instance, learning_rate, indices, values, model, momentum);
}

#define INSTANTIATE_INDEXED_SLICES_MOMENTUM_MODEL_UPDATE_KERNEL_UTIL_CUDA(                 \
    val_type_pair, key_type_pair, idx_type_pair)                                           \
  template struct IndexedSlicesMomentumMdUpdateKernelUtil<                                 \
      DeviceType::kCUDA, OF_PP_PAIR_FIRST(val_type_pair), OF_PP_PAIR_FIRST(key_type_pair), \
      OF_PP_PAIR_FIRST(idx_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_INDEXED_SLICES_MOMENTUM_MODEL_UPDATE_KERNEL_UTIL_CUDA,
                                 FLOATING_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ, INT_DATA_TYPE_SEQ);
#undef INSTANTIATE_INDEXED_SLICES_MOMENTUM_MODEL_UPDATE_KERNEL_UTIL_CUDA

namespace {

__global__ void BiasCorrectionFactorKernelGpu(float beta, const int64_t* train_step, float* out) {
  const auto exponent = static_cast<double>(*train_step + 1);
  const float bias_correction_factor = 1.0 - static_cast<float>(pow(beta, exponent));
  *out = bias_correction_factor;
}

template<typename T, typename G, typename C>
__global__ void AdamUpdateGpu(int64_t n, T scale, float l1, float l2, float beta1, float beta2,
                              float epsilon, float weight_decay, bool amsgrad,
                              bool do_bias_correction, float learning_rate_val, float lr_scale,
                              float bias_correction1_val, float bias_correction2_val,
                              const float* learning_rate, const T* scale_by_ptr,
                              const int64_t* skip_if, const float* bias_correction1_ptr,
                              const float* bias_correction2_ptr, const G* model_diff, T* model,
                              C* model_copy, T* m, T* v, T* max_v) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
  if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
  if (bias_correction1_ptr != nullptr) { bias_correction1_val = *bias_correction1_ptr; }
  if (bias_correction2_ptr != nullptr) { bias_correction2_val = *bias_correction2_ptr; }

  learning_rate_val *= lr_scale;
  CUDA_1D_KERNEL_LOOP(i, n) {
    if (model_copy != nullptr) {
      FusedAdamUpdateFunctor<T, G, C>()(model_diff + i, model + i, model_copy + i, m + i, v + i,
                                        max_v + i, scale, l1, l2, beta1, beta2, epsilon,
                                        weight_decay, amsgrad, bias_correction1_val,
                                        bias_correction2_val, learning_rate_val);
    } else {
      AdamUpdateFunctor<T, G>()(model_diff + i, model + i, m + i, v + i, max_v + i, scale, l1, l2,
                                beta1, beta2, epsilon, weight_decay, amsgrad, bias_correction1_val,
                                bias_correction2_val, learning_rate_val);
    }
  }
}

template<typename T>
__global__ void AdamUpdateBetaTGpu(const T beta1, const T beta2, const int64_t* skip_if, T* beta1_t,
                                   T* beta2_t) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  *beta1_t *= beta1;
  *beta2_t *= beta2;
}

template<typename T, typename K, typename IDX>
__global__ void IndexedSlicesAdamUpdateGpu(
    float beta1, float beta2, float epsilon, float weight_decay, bool amsgrad,
    bool do_bias_correction, float lr, float lr_scale, int64_t feature_size, int64_t lower_bound,
    int64_t upper_bound, const IDX* num_unique_instance, const float* learning_rate,
    const float* bias_correction1_ptr, const float* bias_correction2_ptr, const K* indices,
    const T* values, T* model, T* m, T* v, T* max_v) {
  if (learning_rate != nullptr) { lr = *learning_rate; }
  lr *= lr_scale;
  float bias_correction1 = 1.0;
  float bias_correction2 = 1.0;
  if (bias_correction1_ptr != nullptr) { bias_correction1 = *bias_correction1_ptr; }
  if (bias_correction2_ptr != nullptr) { bias_correction2 = *bias_correction2_ptr; }
  const int64_t n = *num_unique_instance * feature_size;
  CUDA_1D_KERNEL_LOOP(i, n) {
    const IDX indices_idx = i / feature_size;
    const IDX inner_idx = i - indices_idx * feature_size;
    const IDX instance_id = indices[indices_idx];
    if (instance_id >= lower_bound && instance_id < upper_bound) {
      const IDX model_idx = (instance_id - lower_bound) * feature_size + inner_idx;
      AdamUpdateFunctor<T, T>()(values + i, model + model_idx, m + model_idx, v + model_idx,
                                max_v + i, static_cast<T>(1), 0, 0, beta1, beta2, epsilon,
                                weight_decay, amsgrad, bias_correction1, bias_correction2, lr);
    }
  }
}

template<typename T, typename G>
__global__ void LambGradGpu(int64_t n, T scale, float l1, float l2, float beta1, float beta2,
                            float epsilon, const T* scale_by_ptr, const int64_t* skip_if,
                            const G* model_diff, T* adam_diff, T* model, T* m, T* v,
                            bool do_bias_correction, float bias_correction1_val,
                            float bias_correction2_val, const float* bias_correction1_ptr,
                            const float* bias_correction2_ptr) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
  if (bias_correction1_ptr != nullptr) { bias_correction1_val = *bias_correction1_ptr; }
  if (bias_correction2_ptr != nullptr) { bias_correction2_val = *bias_correction2_ptr; }
  CUDA_1D_KERNEL_LOOP(i, n) {
    LambGradFunctor<T, G>()(model_diff + i, adam_diff + i, model + i, m + i, v + i, scale, l1, l2,
                            beta1, beta2, epsilon, do_bias_correction, bias_correction1_val,
                            bias_correction2_val);
  }
}

template<typename T>
__global__ void LambUpdateGpu(int64_t n, float weight_decay, float learning_rate_val,
                              float lr_scale, const float* learning_rate_ptr,
                              const int64_t* skip_if, const T* w_norm_2, const T* g_norm_2,
                              const T* adam_diff, T* model) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (learning_rate_ptr != nullptr) { learning_rate_val = *learning_rate_ptr; }
  learning_rate_val *= lr_scale;
  const float lr = LambLRFunctor<T>()(learning_rate_val, w_norm_2, g_norm_2);
  CUDA_1D_KERNEL_LOOP(i, n) { LambUpdateFunctor<T>()(lr, weight_decay, adam_diff + i, model + i); }
}

}  // namespace

template<typename T, typename G, typename C>
struct AdamUpdateKernelUtil<DeviceType::kCUDA, T, G, C> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float beta1,
                     float beta2, float epsilon, float weight_decay, bool amsgrad,
                     bool do_bias_correction, float learning_rate_val, float lr_scale,
                     float bias_correction1_val, float bias_correction2_val,
                     const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if,
                     const float* bias_correction1_ptr, const float* bias_correction2_ptr,
                     const G* model_diff, T* model, C* model_copy, T* m, T* v, T* max_v);
};

template<typename T, typename G, typename C>
void AdamUpdateKernelUtil<DeviceType::kCUDA, T, G, C>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float beta1, float beta2,
    float epsilon, float weight_decay, bool amsgrad, bool do_bias_correction,
    float learning_rate_val, float lr_scale, float bias_correction1_val, float bias_correction2_val,
    const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if,
    const float* bias_correction1_ptr, const float* bias_correction2_ptr, const G* model_diff,
    T* model, C* model_copy, T* m, T* v, T* max_v) {
  AdamUpdateGpu<T, G, C><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                           stream->As<ep::CudaStream>()->hip_stream()>>>(
      n, scale, l1, l2, beta1, beta2, epsilon, weight_decay, amsgrad, do_bias_correction,
      learning_rate_val, lr_scale, bias_correction1_val, bias_correction2_val, learning_rate,
      scale_by_ptr, skip_if, bias_correction1_ptr, bias_correction2_ptr, model_diff, model,
      model_copy, m, v, max_v);
}

template<typename T, typename G>
struct AdamUpdateKernelUtil<DeviceType::kCUDA, T, G, float16> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float beta1,
                     float beta2, float epsilon, float weight_decay, bool amsgrad,
                     bool do_bias_correction, float learning_rate_val, float lr_scale,
                     float bias_correction1_val, float bias_correction2_val,
                     const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if,
                     const float* bias_correction1_ptr, const float* bias_correction2_ptr,
                     const G* model_diff, T* model, float16* model_copy, T* m, T* v, T* max_v);
};

template<typename T, typename G>
void AdamUpdateKernelUtil<DeviceType::kCUDA, T, G, float16>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float beta1, float beta2,
    float epsilon, float weight_decay, bool amsgrad, bool do_bias_correction,
    float learning_rate_val, float lr_scale, float bias_correction1_val, float bias_correction2_val,
    const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if,
    const float* bias_correction1_ptr, const float* bias_correction2_ptr, const G* model_diff,
    T* model, float16* model_copy, T* m, T* v, T* max_v) {
  AdamUpdateKernelUtil<DeviceType::kCUDA, T, G, half>::Update(
      stream, n, scale, l1, l2, beta1, beta2, epsilon, weight_decay, amsgrad, do_bias_correction,
      learning_rate_val, lr_scale, bias_correction1_val, bias_correction2_val, learning_rate,
      scale_by_ptr, skip_if, bias_correction1_ptr, bias_correction2_ptr, model_diff, model,
      reinterpret_cast<half*>(model_copy), m, v, max_v);
}

template<typename T>
struct AdamUpdateKernelUtil<DeviceType::kCUDA, T, float16, float16> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float beta1,
                     float beta2, float epsilon, float weight_decay, bool amsgrad,
                     bool do_bias_correction, float learning_rate_val, float lr_scale,
                     float bias_correction1_val, float bias_correction2_val,
                     const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if,
                     const float* bias_correction1_ptr, const float* bias_correction2_ptr,
                     const float16* model_diff, T* model, float16* model_copy, T* m, T* v,
                     T* max_v);
};

template<typename T>
void AdamUpdateKernelUtil<DeviceType::kCUDA, T, float16, float16>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float beta1, float beta2,
    float epsilon, float weight_decay, bool amsgrad, bool do_bias_correction,
    float learning_rate_val, float lr_scale, float bias_correction1_val, float bias_correction2_val,
    const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if,
    const float* bias_correction1_ptr, const float* bias_correction2_ptr, const float16* model_diff,
    T* model, float16* model_copy, T* m, T* v, T* max_v) {
  AdamUpdateKernelUtil<DeviceType::kCUDA, T, half, half>::Update(
      stream, n, scale, l1, l2, beta1, beta2, epsilon, weight_decay, amsgrad, do_bias_correction,
      learning_rate_val, lr_scale, bias_correction1_val, bias_correction2_val, learning_rate,
      scale_by_ptr, skip_if, bias_correction1_ptr, bias_correction2_ptr,
      reinterpret_cast<const half*>(model_diff), model, reinterpret_cast<half*>(model_copy), m, v,
      max_v);
}

template struct AdamUpdateKernelUtil<DeviceType::kCUDA, float, float, float16>;
template struct AdamUpdateKernelUtil<DeviceType::kCUDA, double, double, float16>;
template struct AdamUpdateKernelUtil<DeviceType::kCUDA, float, float16, float16>;

template<typename T, typename G>
__global__ void AdagradUpdateGpu(int64_t n, T scale, float l1, float l2, float lr_decay,
                                 float epsilon, float weight_decay, float learning_rate_val,
                                 float lr_scale, int64_t train_step, const float* learning_rate,
                                 const int64_t* train_step_ptr, const T* scale_by_ptr,
                                 const int64_t* skip_if, const G* model_diff, T* model, T* sum) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
  if (train_step_ptr != nullptr) {
    train_step = *train_step_ptr + 1;
  }  // train_step_ptr start from zero.
  if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
  learning_rate_val = learning_rate_val * lr_scale / (1 + (train_step - 1) * lr_decay);
  CUDA_1D_KERNEL_LOOP(i, n) {
    AdagradUpdateFunctor<T, G>()(model_diff + i, model + i, sum + i, scale, l1, l2, epsilon,
                                 weight_decay, learning_rate_val);
  }
}

template<typename T, typename G>
struct AdagradUpdateKernelUtil<DeviceType::kCUDA, T, G> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float lr_decay,
                     float epsilon, float weight_decay, float learning_rate_val, float lr_scale,
                     int64_t train_step, const float* learning_rate, const int64_t* train_step_ptr,
                     const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff, T* model,
                     T* sum);
};

template<typename T, typename G>
void AdagradUpdateKernelUtil<DeviceType::kCUDA, T, G>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float lr_decay, float epsilon,
    float weight_decay, float learning_rate_val, float lr_scale, int64_t train_step,
    const float* learning_rate, const int64_t* train_step_ptr, const T* scale_by_ptr,
    const int64_t* skip_if, const G* model_diff, T* model, T* sum) {
  AdagradUpdateGpu<T, G><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                           stream->As<ep::CudaStream>()->hip_stream()>>>(
      n, scale, l1, l2, lr_decay, epsilon, weight_decay, learning_rate_val, lr_scale, train_step,
      learning_rate, train_step_ptr, scale_by_ptr, skip_if, model_diff, model, sum);
}

template struct AdagradUpdateKernelUtil<DeviceType::kCUDA, float, float>;
template struct AdagradUpdateKernelUtil<DeviceType::kCUDA, double, double>;

template<typename T, typename G>
struct LambUpdateKernelUtil<DeviceType::kCUDA, T, G> {
  static void Update(ep::Stream* stream, int64_t n, float scale, float l1, float l2, float beta1,
                     float beta2, float epsilon, float weight_decay, float learning_rate_val,
                     float lr_scale, bool do_bias_correction, float bias_correction1_val,
                     float bias_correction2_val, const float* learning_rate_ptr,
                     const float* bias_correction1_ptr, const float* bias_correction2_ptr,
                     const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff,
                     T* adam_diff, T* model, T* m, T* v, T* norm_buffer);
};

template<typename T, typename G>
void LambUpdateKernelUtil<DeviceType::kCUDA, T, G>::Update(
    ep::Stream* stream, int64_t n, float scale, float l1, float l2, float beta1, float beta2,
    float epsilon, float weight_decay, float learning_rate_val, float lr_scale,
    bool do_bias_correction, float bias_correction1_val, float bias_correction2_val,
    const float* learning_rate_ptr, const float* bias_correction1_ptr,
    const float* bias_correction2_ptr, const T* scale_by_ptr, const int64_t* skip_if,
    const G* model_diff, T* adam_diff, T* model, T* m, T* v, T* norm_buffer) {
  LambGradGpu<T, G><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                      stream->As<ep::CudaStream>()->hip_stream()>>>(
      n, scale, l1, l2, beta1, beta2, epsilon, scale_by_ptr, skip_if, model_diff, adam_diff, model,
      m, v, do_bias_correction, bias_correction1_val, bias_correction2_val, bias_correction1_ptr,
      bias_correction2_ptr);
  T* w_norm_2 = norm_buffer;
  T* g_norm_2 = norm_buffer + 1;
  Memset<DeviceType::kCUDA>(stream, norm_buffer, 0, 2 * sizeof(T));
  SumSquares2<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
         stream->As<ep::CudaStream>()->hip_stream()>>>(n, model, w_norm_2, adam_diff, g_norm_2);
  LambUpdateGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                     stream->As<ep::CudaStream>()->hip_stream()>>>(
      n, weight_decay, learning_rate_val, lr_scale, learning_rate_ptr, skip_if, w_norm_2, g_norm_2,
      adam_diff, model);
}

template<typename T>
struct LambUpdateKernelUtil<DeviceType::kCUDA, T, float16> {
  static void Update(ep::Stream* stream, int64_t n, float scale, float l1, float l2, float beta1,
                     float beta2, float epsilon, float weight_decay, float learning_rate_val,
                     float lr_scale, bool do_bias_correction, float bias_correction1_val,
                     float bias_correction2_val, const float* learning_rate_ptr,
                     const float* bias_correction1_ptr, const float* bias_correction2_ptr,
                     const T* scale_by_ptr, const int64_t* skip_if, const float16* model_diff,
                     T* adam_diff, T* model, T* m, T* v, T* norm_buffer);
};

template<typename T>
void LambUpdateKernelUtil<DeviceType::kCUDA, T, float16>::Update(
    ep::Stream* stream, int64_t n, float scale, float l1, float l2, float beta1, float beta2,
    float epsilon, float weight_decay, float learning_rate_val, float lr_scale,
    bool do_bias_correction, float bias_correction1_val, float bias_correction2_val,
    const float* learning_rate_ptr, const float* bias_correction1_ptr,
    const float* bias_correction2_ptr, const T* scale_by_ptr, const int64_t* skip_if,
    const float16* model_diff, T* adam_diff, T* model, T* m, T* v, T* norm_buffer) {
  LambUpdateKernelUtil<DeviceType::kCUDA, T, half>::Update(
      stream, n, scale, l1, l2, beta1, beta2, epsilon, weight_decay, learning_rate_val, lr_scale,
      do_bias_correction, bias_correction1_val, bias_correction2_val, learning_rate_ptr,
      bias_correction1_ptr, bias_correction2_ptr, scale_by_ptr, skip_if,
      reinterpret_cast<const half*>(model_diff), adam_diff, model, m, v, norm_buffer);
}

template struct LambUpdateKernelUtil<DeviceType::kCUDA, float, float>;
template struct LambUpdateKernelUtil<DeviceType::kCUDA, double, double>;
template struct LambUpdateKernelUtil<DeviceType::kCUDA, float, float16>;

template<typename T, typename K, typename IDX>
struct IndexedSlicesAdamMdUpdateKernelUtil<DeviceType::kCUDA, T, K, IDX> {
  static void Update(ep::Stream* stream, float beta1, float beta2, float epsilon,
                     float weight_decay, bool amsgrad, bool do_bias_correction, float lr,
                     float lr_scale, int64_t num_instance, int64_t feature_size,
                     int64_t lower_bound, int64_t upper_bound, const IDX* num_unique_instance,
                     const float* learning_rate, const float* bias_correction1_ptr,
                     const float* bias_correction2_ptr, const K* indices, const T* values, T* model,
                     T* m, T* v, T* max_v);
};

template<typename T, typename K, typename IDX>
void IndexedSlicesAdamMdUpdateKernelUtil<DeviceType::kCUDA, T, K, IDX>::Update(
    ep::Stream* stream, float beta1, float beta2, float epsilon, float weight_decay, bool amsgrad,
    bool do_bias_correction, float lr, float lr_scale, int64_t num_instance, int64_t feature_size,
    int64_t lower_bound, int64_t upper_bound, const IDX* num_unique_instance,
    const float* learning_rate, const float* bias_correction1_ptr,
    const float* bias_correction2_ptr, const K* indices, const T* values, T* model, T* m, T* v,
    T* max_v) {
  IndexedSlicesAdamUpdateGpu<T, K, IDX>
      <<<BlocksNum4ThreadsNum(num_instance * feature_size), kCudaThreadsNumPerBlock, 0,
         stream->As<ep::CudaStream>()->hip_stream()>>>(
          beta1, beta2, epsilon, weight_decay, amsgrad, do_bias_correction, lr, lr_scale,
          feature_size, lower_bound, upper_bound, num_unique_instance, learning_rate,
          bias_correction1_ptr, bias_correction2_ptr, indices, values, model, m, v, max_v);
}
#define INSTANTIATE_INDEXED_SLICES_ADAM_MODEL_UPDATE_KERNEL_UTIL_CUDA(                     \
    val_type_pair, key_type_pair, idx_type_pair)                                           \
  template struct IndexedSlicesAdamMdUpdateKernelUtil<                                     \
      DeviceType::kCUDA, OF_PP_PAIR_FIRST(val_type_pair), OF_PP_PAIR_FIRST(key_type_pair), \
      OF_PP_PAIR_FIRST(idx_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_INDEXED_SLICES_ADAM_MODEL_UPDATE_KERNEL_UTIL_CUDA,
                                 FLOATING_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ, INT_DATA_TYPE_SEQ);
#undef INSTANTIATE_INDEXED_SLICES_ADAM_MODEL_UPDATE_KERNEL_UTIL_CUDA

template<>
struct BiasCorrectionFactorKernelUtil<DeviceType::kCUDA> {
  static void BiasCorrectionFactorCompute(ep::Stream* stream, float beta, const int64_t* train_step,
                                          float* out);
};

void BiasCorrectionFactorKernelUtil<DeviceType::kCUDA>::BiasCorrectionFactorCompute(
    ep::Stream* stream, float beta, const int64_t* train_step, float* out) {
  BiasCorrectionFactorKernelGpu<<<1, 1, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
      beta, train_step, out);
}

namespace {

template<typename T, typename G, bool centered>
__global__ void RmsPropUpdateGpu(int64_t n, T scale, float l1, float l2, T* mean_square,
                                 T* mean_gradient, float epsilon, float weight_decay,
                                 float decay_rate, float learning_rate_val, float lr_scale,
                                 const float* learning_rate, const T* scale_by_ptr,
                                 const int64_t* skip_if, const G* model_diff, T* model) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
  if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
  learning_rate_val *= lr_scale;
  CUDA_1D_KERNEL_LOOP(i, n) {
    RmsPropUpdateFunctor<T, G, centered>()(model_diff + i, model + i, n, scale, l1, l2,
                                           mean_square + i,
                                           (centered ? mean_gradient + i : nullptr), epsilon,
                                           weight_decay, decay_rate, learning_rate_val);
  }
}

}  // namespace

template<typename T, typename G>
struct RmsPropUpdateKernelUtil<DeviceType::kCUDA, T, G> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, bool centered,
                     float epsilon, float weight_decay, float decay_rate, float learning_rate_val,
                     float lr_scale, const float* learning_rate, const T* scale_by_ptr,
                     const int64_t* skip_if, const G* model_diff, T* model, T* mean_square,
                     T* mean_gradient);
};

template<typename T, typename G>
void RmsPropUpdateKernelUtil<DeviceType::kCUDA, T, G>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, bool centered, float epsilon,
    float weight_decay, float decay_rate, float learning_rate_val, float lr_scale,
    const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff,
    T* model, T* mean_square, T* mean_gradient) {
  if (centered) {
    RmsPropUpdateGpu<T, G, true><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                                   stream->As<ep::CudaStream>()->hip_stream()>>>(
        n, scale, l1, l2, mean_square, mean_gradient, epsilon, weight_decay, decay_rate,
        learning_rate_val, lr_scale, learning_rate, scale_by_ptr, skip_if, model_diff, model);
  } else {
    RmsPropUpdateGpu<T, G, false><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                                    stream->As<ep::CudaStream>()->hip_stream()>>>(
        n, scale, l1, l2, mean_square, mean_gradient, epsilon, weight_decay, decay_rate,
        learning_rate_val, lr_scale, learning_rate, scale_by_ptr, skip_if, model_diff, model);
  }
}

template<typename T>
struct RmsPropUpdateKernelUtil<DeviceType::kCUDA, T, float16> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, bool centered,
                     float epsilon, float weight_decay, float decay_rate, float learning_rate_val,
                     float lr_scale, const float* learning_rate, const T* scale_by_ptr,
                     const int64_t* skip_if, const float16* model_diff, T* model, T* mean_square,
                     T* mean_gradient);
};

template<typename T>
void RmsPropUpdateKernelUtil<DeviceType::kCUDA, T, float16>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, bool centered, float epsilon,
    float weight_decay, float decay_rate, float learning_rate_val, float lr_scale,
    const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if,
    const float16* model_diff, T* model, T* mean_square, T* mean_gradient) {
  RmsPropUpdateKernelUtil<DeviceType::kCUDA, T, half>::Update(
      stream, n, scale, l1, l2, centered, epsilon, weight_decay, decay_rate, learning_rate_val,
      lr_scale, learning_rate, scale_by_ptr, skip_if, reinterpret_cast<const half*>(model_diff),
      model, mean_square, mean_gradient);
}

template struct RmsPropUpdateKernelUtil<DeviceType::kCUDA, float, float>;
template struct RmsPropUpdateKernelUtil<DeviceType::kCUDA, double, double>;
template struct RmsPropUpdateKernelUtil<DeviceType::kCUDA, float, float16>;

namespace {

template<typename T, typename G>
__global__ void LarsScaleModelDiffGpu(int64_t n, T scale, float l1, float l2, const T* scale_by_ptr,
                                      const int64_t* skip_if, const G* model_diff, T* model,
                                      T* model_diff_tmp) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
  CUDA_1D_KERNEL_LOOP(i, n) {
    model_diff_tmp[i] =
        CastScaleRegularizeGradientFunctor<T, G>()(model_diff[i], model[i], scale, l1, l2);
  }
}

template<typename T>
__global__ void LarsGetLocalLearningRateGpu(const float* learning_rate, float lr_scale,
                                            T weight_decay, T epsilon, T lars_coefficient,
                                            const int64_t* skip_if, T* data_tmp) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  T* model_norm = &data_tmp[0];
  T* model_diff_norm = &data_tmp[1];
  T* local_learning_rate = &data_tmp[2];
  *model_norm = std::sqrt(*model_norm);
  *model_diff_norm = std::sqrt(*model_diff_norm);
  T lars = static_cast<T>(1);
  if (*model_norm > 0 && *model_diff_norm > 0) {
    lars = lars_coefficient * (*model_norm)
           / (epsilon + (*model_diff_norm) + weight_decay * (*model_norm));
  }
  T lr = *learning_rate;
  lr *= lr_scale;
  *local_learning_rate = lr * lars;
}

template<typename T>
__global__ void LarsUpdateGpu(int64_t n, float momentum_beta, T* momentum, float weight_decay,
                              const int64_t* skip_if, T* local_learning_rate, T* model_diff_tmp,
                              T* model) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  CUDA_1D_KERNEL_LOOP(i, n) {
    LarsUpdateFunctor<T>()(model_diff_tmp + i, model + i, momentum_beta, momentum + i, weight_decay,
                           *local_learning_rate);
  }
}

}  // namespace

template<typename T, typename G>
struct LarsUpdateKernelUtil<DeviceType::kCUDA, T, G> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2,
                     float momentum_beta, float epsilon, float lars_coefficient, float weight_decay,
                     float lr_scale, const float* learning_rate, const T* scale_by_ptr,
                     const int64_t* skip_if, const G* model_diff, T* model, T* momentum,
                     T* data_tmp, T* model_diff_tmp);
};

template<typename T, typename G>
void LarsUpdateKernelUtil<DeviceType::kCUDA, T, G>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float momentum_beta, float epsilon,
    float lars_coefficient, float weight_decay, float lr_scale, const float* learning_rate,
    const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff, T* model, T* momentum,
    T* data_tmp, T* model_diff_tmp) {
  LarsScaleModelDiffGpu<T, G><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                                stream->As<ep::CudaStream>()->hip_stream()>>>(
      n, scale, l1, l2, scale_by_ptr, skip_if, model_diff, model, model_diff_tmp);
  T* model_norm = data_tmp;
  T* model_diff_norm = data_tmp + 1;
  T* local_learning_rate = data_tmp + 2;
  Memset<DeviceType::kCUDA>(stream, data_tmp, 0, 2 * sizeof(T));
  SumSquares2<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                   stream->As<ep::CudaStream>()->hip_stream()>>>(n, model, model_norm,
                                                                  model_diff_tmp, model_diff_norm);
  LarsGetLocalLearningRateGpu<T><<<1, 1, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
      learning_rate, lr_scale, weight_decay, epsilon, lars_coefficient, skip_if, data_tmp);
  LarsUpdateGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                     stream->As<ep::CudaStream>()->hip_stream()>>>(
      n, momentum_beta, momentum, weight_decay, skip_if, local_learning_rate, model_diff_tmp,
      model);
}

template<typename T>
struct LarsUpdateKernelUtil<DeviceType::kCUDA, T, float16> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2,
                     float momentum_beta, float epsilon, float lars_coefficient, float weight_decay,
                     float lr_scale, const float* learning_rate, const T* scale_by_ptr,
                     const int64_t* skip_if, const float16* model_diff, T* model, T* momentum,
                     T* data_tmp, T* model_diff_tmp);
};

template<typename T>
void LarsUpdateKernelUtil<DeviceType::kCUDA, T, float16>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float momentum_beta, float epsilon,
    float lars_coefficient, float weight_decay, float lr_scale, const float* learning_rate,
    const T* scale_by_ptr, const int64_t* skip_if, const float16* model_diff, T* model, T* momentum,
    T* data_tmp, T* model_diff_tmp) {
  LarsUpdateKernelUtil<DeviceType::kCUDA, T, half>::Update(
      stream, n, scale, l1, l2, momentum_beta, epsilon, lars_coefficient, weight_decay, lr_scale,
      learning_rate, scale_by_ptr, skip_if, reinterpret_cast<const half*>(model_diff), model,
      momentum, data_tmp, model_diff_tmp);
}

template struct LarsUpdateKernelUtil<DeviceType::kCUDA, float, float>;
template struct LarsUpdateKernelUtil<DeviceType::kCUDA, double, double>;
template struct LarsUpdateKernelUtil<DeviceType::kCUDA, float, float16>;

template<typename T, typename G>
__global__ void FtrlUpdateGpu(int64_t n, T scale, float l1, float l2, float lr_power, float lambda1,
                              float lambda2, float beta, float weight_decay,
                              float learning_rate_val, float lr_scale, const float* learning_rate,
                              const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff,
                              T* model, T* accumulate, T* z) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
  if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
  learning_rate_val *= lr_scale;
  CUDA_1D_KERNEL_LOOP(i, n) {
    FtrlUpdateFunctor<T, G>()(model_diff + i, model + i, accumulate + i, z + i, scale, l1, l2,
                              lr_power, lambda1, lambda2, beta, weight_decay, learning_rate_val);
  }
}

template<typename T, typename G>
struct FtrlUpdateKernelUtil<DeviceType::kCUDA, T, G> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float lr_power,
                     float lambda1, float lambda2, float beta, float weight_decay,
                     float learning_rate_val, float lr_scale, const float* learning_rate,
                     const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff, T* model,
                     T* accumulate, T* z);
};

template<typename T, typename G>
void FtrlUpdateKernelUtil<DeviceType::kCUDA, T, G>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float lr_power, float lambda1,
    float lambda2, float beta, float weight_decay, float learning_rate_val, float lr_scale,
    const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff,
    T* model, T* accumulate, T* z) {
  FtrlUpdateGpu<T, G><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                        stream->As<ep::CudaStream>()->hip_stream()>>>(
      n, scale, l1, l2, lr_power, lambda1, lambda2, beta, weight_decay, learning_rate_val, lr_scale,
      learning_rate, scale_by_ptr, skip_if, model_diff, model, accumulate, z);
}

template<typename T>
struct FtrlUpdateKernelUtil<DeviceType::kCUDA, T, float16> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float lr_power,
                     float lambda1, float lambda2, float beta, float weight_decay,
                     float learning_rate_val, float lr_scale, const float* learning_rate,
                     const T* scale_by_ptr, const int64_t* skip_if, const float16* model_diff,
                     T* model, T* accumulate, T* z);
};

template<typename T>
void FtrlUpdateKernelUtil<DeviceType::kCUDA, T, float16>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float lr_power, float lambda1,
    float lambda2, float beta, float weight_decay, float learning_rate_val, float lr_scale,
    const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if,
    const float16* model_diff, T* model, T* accumulate, T* z) {
  FtrlUpdateKernelUtil<DeviceType::kCUDA, T, half>::Update(
      stream, n, scale, l1, l2, lr_power, lambda1, lambda2, beta, weight_decay, learning_rate_val,
      lr_scale, learning_rate, scale_by_ptr, skip_if, reinterpret_cast<const half*>(model_diff),
      model, accumulate, z);
}

template struct FtrlUpdateKernelUtil<DeviceType::kCUDA, float, float>;
template struct FtrlUpdateKernelUtil<DeviceType::kCUDA, double, double>;
template struct FtrlUpdateKernelUtil<DeviceType::kCUDA, float, float16>;

template<typename T, typename G>
__global__ void AdadeltaUpdateGpu(int64_t n, T scale, float l1, float l2, float rho, float epsilon,
                                  bool maximize, float weight_decay, float learning_rate_val,
                                  float lr_scale, const float* learning_rate, const T* scale_by_ptr,
                                  const int64_t* skip_if, const G* model_diff, T* model,
                                  T* square_avgs, T* acc_deltas) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
  if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
  learning_rate_val *= lr_scale;
  CUDA_1D_KERNEL_LOOP(i, n) {
    AdadeltaUpdateFunctor<T, G>()(model_diff + i, model + i, square_avgs + i, acc_deltas + i, scale,
                                  l1, l2, rho, epsilon, maximize, weight_decay, learning_rate_val);
  }
}

template<typename T, typename G>
struct AdadeltaUpdateKernelUtil<DeviceType::kCUDA, T, G> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float rho,
                     float epsilon, bool maximize, float weight_decay, float learning_rate_val,
                     float lr_scale, const float* learning_rate, const T* scale_by_ptr,
                     const int64_t* skip_if, const G* model_diff, T* model, T* square_avgs,
                     T* acc_deltas);
};

template<typename T, typename G>
void AdadeltaUpdateKernelUtil<DeviceType::kCUDA, T, G>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float rho, float epsilon,
    bool maximize, float weight_decay, float learning_rate_val, float lr_scale,
    const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if, const G* model_diff,
    T* model, T* square_avgs, T* acc_deltas) {
  AdadeltaUpdateGpu<T, G><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                            stream->As<ep::CudaStream>()->hip_stream()>>>(
      n, scale, l1, l2, rho, epsilon, maximize, weight_decay, learning_rate_val, lr_scale,
      learning_rate, scale_by_ptr, skip_if, model_diff, model, square_avgs, acc_deltas);
}

template<typename T>
struct AdadeltaUpdateKernelUtil<DeviceType::kCUDA, T, float16> {
  static void Update(ep::Stream* stream, int64_t n, T scale, float l1, float l2, float rho,
                     float epsilon, bool maximize, float weight_decay, float learning_rate_val,
                     float lr_scale, const float* learning_rate, const T* scale_by_ptr,
                     const int64_t* skip_if, const float16* model_diff, T* model, T* square_avgs,
                     T* acc_deltas);
};

template<typename T>
void AdadeltaUpdateKernelUtil<DeviceType::kCUDA, T, float16>::Update(
    ep::Stream* stream, int64_t n, T scale, float l1, float l2, float rho, float epsilon,
    bool maximize, float weight_decay, float learning_rate_val, float lr_scale,
    const float* learning_rate, const T* scale_by_ptr, const int64_t* skip_if,
    const float16* model_diff, T* model, T* square_avgs, T* acc_deltas) {
  AdadeltaUpdateKernelUtil<DeviceType::kCUDA, T, half>::Update(
      stream, n, scale, l1, l2, rho, epsilon, maximize, weight_decay, learning_rate_val, lr_scale,
      learning_rate, scale_by_ptr, skip_if, reinterpret_cast<const half*>(model_diff), model,
      square_avgs, acc_deltas);
}

template struct AdadeltaUpdateKernelUtil<DeviceType::kCUDA, float, float>;
template struct AdadeltaUpdateKernelUtil<DeviceType::kCUDA, double, double>;
template struct AdadeltaUpdateKernelUtil<DeviceType::kCUDA, float, float16>;

}  // namespace oneflow
