#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/nd_index_slice_kernels.h"
#include "oneflow/core/cuda/atomic.cuh"

namespace oneflow {

namespace {

template<typename T, typename I>
__global__ void CudaGatherNd(NdIndexSliceArgs args, const I* indices, const T* dense, T* slices) {
  DoGatherNd(args.num_slices * args.slice_size, args.slice_size, args.index_ndims, args.dense_shape,
             indices, dense, slices);
}

template<typename T, typename I>
__global__ void CudaScatterNdAdd(NdIndexSliceArgs args, const I* indices, const T* slices,
                                 T* dense) {
  DoScatterNdAdd<DeviceType::kCUDA>(args.num_slices * args.slice_size, args.slice_size,
                                    args.index_ndims, args.dense_shape, indices, slices, dense);
}

template<typename T, typename I>
__global__ void CudaScatterNdUpdate(NdIndexSliceArgs args, const I* indices, const T* slices,
                                    T* dense) {
  DoScatterNdUpdate<DeviceType::kCUDA>(args.num_slices * args.slice_size, args.slice_size,
                                       args.index_ndims, args.dense_shape, indices, slices, dense);
}

template<typename T, typename I>
__global__ void CudaScatterNdUpdateWithStride(NdIndexSliceArgs args, const I* indices,
                                              const T* slices, T* dense) {
  DoScatterNdUpdateWithStride<DeviceType::kCUDA>(args.num_slices * args.slice_size, args, indices,
                                                 slices, dense);
}

template<typename T, typename I>
__global__ void CudaFillByNdIndex(NdIndexSliceArgs args, const I* indices, T* dense, T value) {
  DoFillByNdIndex(args.num_slices * args.slice_size, args.slice_size, args.index_ndims,
                  args.dense_shape, indices, dense, value);
}

}  // namespace

template<typename T, typename I>
struct GatherNdFunctor<DeviceType::kCUDA, T, I> final {
  void operator()(ep::Stream* stream, const NdIndexSliceArgs& args, const I* indices,
                  const T* dense, T* slices) const {
    RUN_CUDA_KERNEL((CudaGatherNd<T, I>), stream, args.num_slices * args.slice_size, args, indices,
                    dense, slices);
  }
};

template<typename T, typename I>
struct ScatterNdAddFunctor<DeviceType::kCUDA, T, I> final {
  void operator()(ep::Stream* stream, const NdIndexSliceArgs& args, const I* indices,
                  const T* slices, T* dense) const {
    RUN_CUDA_KERNEL((CudaScatterNdAdd<T, I>), stream, args.num_slices * args.slice_size, args,
                    indices, slices, dense);
  }
};

template<typename T, typename I>
struct ScatterNdUpdateFunctor<DeviceType::kCUDA, T, I> final {
  void operator()(ep::Stream* stream, const NdIndexSliceArgs& args, const I* indices,
                  const T* slices, T* dense) const {
    RUN_CUDA_KERNEL((CudaScatterNdUpdate<T, I>), stream, args.num_slices * args.slice_size, args,
                    indices, slices, dense);
  }
};

template<typename T, typename I>
struct ScatterNdUpdateWithStrideFunctor<DeviceType::kCUDA, T, I> final {
  void operator()(ep::Stream* stream, const NdIndexSliceArgs& args, const I* indices,
                  const T* slices, T* dense) const {
    RUN_CUDA_KERNEL((CudaScatterNdUpdateWithStride<T, I>), stream,
                    args.num_slices * args.slice_size, args, indices, slices, dense);
  }
};

template<typename T, typename I>
struct FillByNdIndexFunctor<DeviceType::kCUDA, T, I> final {
  void operator()(ep::Stream* stream, const NdIndexSliceArgs& args, const I* indices, T* dense,
                  T value) const {
    RUN_CUDA_KERNEL((CudaFillByNdIndex<T, I>), stream, args.num_slices * args.slice_size, args,
                    indices, dense, value);
  }
};

template<typename T>
struct DeviceAdd<DeviceType::kCUDA, T> {
  __device__ __forceinline__ static void Invoke(const T* x, T* y) { cuda::atomic::Add(y, *x); }
};

template<>
struct DeviceAdd<DeviceType::kCUDA, bool> {
  __device__ __forceinline__ static void Invoke(const bool* x, bool* y) { *y += *x; }
};

template<>
struct DeviceAdd<DeviceType::kCUDA, uint8_t> {
  __device__ __forceinline__ static void Invoke(const uint8_t* x, uint8_t* y) { *y += *x; }
};

template<>
struct DeviceAdd<DeviceType::kCUDA, int8_t> {
  __device__ __forceinline__ static void Invoke(const int8_t* x, int8_t* y) { *y += *x; }
};

template<>
struct DeviceAdd<DeviceType::kCUDA, int64_t> {
  __device__ __forceinline__ static void Invoke(const int64_t* x, int64_t* y) { *y += *x; }
};

#define CUDA_ATOMIC_ADD_SUPPORTED_DATA_TYPE_SEQ \
  FLOATING_DATA_TYPE_SEQ                        \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
    INSTANTIATE_GATHER_ND_FUNCTOR, (DeviceType::kCUDA),
    ARITHMETIC_DATA_TYPE_SEQ UNSIGNED_INT_DATA_TYPE_SEQ BOOL_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_SCATTER_ND_ADD_FUNCTOR, (DeviceType::kCUDA),
                                 CUDA_ATOMIC_ADD_SUPPORTED_DATA_TYPE_SEQ BOOL_DATA_TYPE_SEQ,
                                 INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_FILL_BY_ND_INDEX_FUNCTOR, (DeviceType::kCUDA),
                                 ARITHMETIC_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
    REGISTER_GATHER_ND_KERNELS, (DeviceType::kCUDA),
    ARITHMETIC_DATA_TYPE_SEQ UNSIGNED_INT_DATA_TYPE_SEQ BOOL_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
    REGISTER_SCATTER_ND_KERNELS, (DeviceType::kCUDA),
    ARITHMETIC_DATA_TYPE_SEQ UNSIGNED_INT_DATA_TYPE_SEQ BOOL_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_SCATTER_ND_LIKE_KERNELS, (DeviceType::kCUDA),
                                 CUDA_ATOMIC_ADD_SUPPORTED_DATA_TYPE_SEQ BOOL_DATA_TYPE_SEQ,
                                 INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
    REGISTER_TENSOR_GATHER_ND_UPDATE_KERNELS, (DeviceType::kCUDA),
    ARITHMETIC_DATA_TYPE_SEQ UNSIGNED_INT_DATA_TYPE_SEQ BOOL_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_TENSOR_GATHER_ND_ADD_KERNELS, (DeviceType::kCUDA),
                                 CUDA_ATOMIC_ADD_SUPPORTED_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

template<>
struct DeviceAdd<DeviceType::kCUDA, float16> {
  __device__ __forceinline__ static void Invoke(const float16* x, float16* y) {
    cuda::atomic::Add(reinterpret_cast<half*>(y), *(reinterpret_cast<const half*>(x)));
  }
};

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_ND_INDEX_SLICE_FUNCTORS, (DeviceType::kCUDA),
                                 FLOAT16_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_ND_INDEX_SLICE_KERNELS, (DeviceType::kCUDA),
                                 FLOAT16_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

#if defined(__CUDA_BF16_TYPES_EXIST__)
template<>
struct DeviceAdd<DeviceType::kCUDA, bfloat16> {
  __device__ __forceinline__ static void Invoke(const bfloat16* x, bfloat16* y) {
    cuda::atomic::Add(reinterpret_cast<hip_bfloat16*>(y),
                      *(reinterpret_cast<const hip_bfloat16*>(x)));
  }
};
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_ND_INDEX_SLICE_FUNCTORS, (DeviceType::kCUDA),
                                 BFLOAT16_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_ND_INDEX_SLICE_KERNELS, (DeviceType::kCUDA),
                                 BFLOAT16_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)
#endif

}  // namespace oneflow
