#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/nd_index_slice_kernels.h"
#include "oneflow/core/cuda/atomic.cuh"

namespace oneflow {

namespace {

template<typename T, typename I>
__global__ void CudaGatherNd(NdIndexSliceArgs<T, I> args, const I* indices, const T* dense,
                             T* slices) {
  DoGatherNd(args.num_slices * args.slice_size, args.slice_size, args.index_ndims, args.dense_shape,
             indices, dense, slices);
}

template<typename T, typename I>
__global__ void CudaScatterNdAdd(NdIndexSliceArgs<T, I> args, const I* indices, const T* slices,
                                 T* dense) {
  DoScatterNdAdd<DeviceType::kGPU>(args.num_slices * args.slice_size, args.slice_size,
                                   args.index_ndims, args.dense_shape, indices, slices, dense);
}

template<typename T, typename I>
__global__ void CudaZeroByNdIndex(NdIndexSliceArgs<T, I> args, const I* indices, T* dense) {
  DoZeroByNdIndex(args.num_slices * args.slice_size, args.slice_size, args.index_ndims,
                  args.dense_shape, indices, dense);
}

}  // namespace

template<typename T, typename I>
struct GatherNdFunctor<DeviceType::kGPU, T, I> final {
  void operator()(DeviceCtx* ctx, const NdIndexSliceArgs<T, I>& args, const I* indices,
                  const T* dense, T* slices) const {
    RUN_CUDA_KERNEL((CudaGatherNd<T, I>), ctx, args.num_slices * args.slice_size, args, indices,
                    dense, slices);
  }
};

template<typename T, typename I>
struct ScatterNdAddFunctor<DeviceType::kGPU, T, I> final {
  void operator()(DeviceCtx* ctx, const NdIndexSliceArgs<T, I>& args, const I* indices,
                  const T* slices, T* dense) const {
    RUN_CUDA_KERNEL((CudaScatterNdAdd<T, I>), ctx, args.num_slices * args.slice_size, args, indices,
                    slices, dense);
  }
};

template<typename T, typename I>
struct ZeroByNdIndexFunctor<DeviceType::kGPU, T, I> final {
  void operator()(DeviceCtx* ctx, const NdIndexSliceArgs<T, I>& args, const I* indices,
                  T* dense) const {
    RUN_CUDA_KERNEL((CudaZeroByNdIndex<T, I>), ctx, args.num_slices * args.slice_size, args,
                    indices, dense);
  }
};

template<typename T>
struct DeviceAdd<DeviceType::kGPU, T> {
  __device__ __forceinline__ static void Invoke(const T* x, T* y) { cuda::atomic::Add(y, *x); }
};

#define GPU_ATOMIC_ADD_SUPPORTED_DATA_TYPE_SEQ \
  FLOATING_DATA_TYPE_SEQ                       \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_GATHER_ND_FUNCTOR, (DeviceType::kGPU),
                                 ARITHMETIC_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_SCATTER_ND_ADD_FUNCTOR, (DeviceType::kGPU),
                                 GPU_ATOMIC_ADD_SUPPORTED_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_ZERO_BY_ND_INDEX_FUNCTOR, (DeviceType::kGPU),
                                 ARITHMETIC_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_GATHER_ND_KERNELS, (DeviceType::kGPU),
                                 ARITHMETIC_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_SCATTER_ND_KERNELS, (DeviceType::kGPU),
                                 GPU_ATOMIC_ADD_SUPPORTED_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_SCATTER_ND_LIKE_KERNELS, (DeviceType::kGPU),
                                 GPU_ATOMIC_ADD_SUPPORTED_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_TENSOR_GATHER_ND_UPDATE_KERNELS, (DeviceType::kGPU),
                                 GPU_ATOMIC_ADD_SUPPORTED_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_TENSOR_GATHER_ND_ADD_KERNELS, (DeviceType::kGPU),
                                 GPU_ATOMIC_ADD_SUPPORTED_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700 && CUDA_VERSION >= 10000

template<>
struct DeviceAdd<DeviceType::kGPU, float16> {
  __device__ __forceinline__ static void Invoke(const float16* x, float16* y) {
    cuda::atomic::Add(reinterpret_cast<half*>(y), *(reinterpret_cast<const half*>(x)));
  }
};

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_ND_INDEX_SLICE_FUNCTORS, (DeviceType::kGPU),
                                 FLOAT16_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_ND_INDEX_SLICE_KERNELS, (DeviceType::kGPU),
                                 FLOAT16_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

#endif

}  // namespace oneflow
