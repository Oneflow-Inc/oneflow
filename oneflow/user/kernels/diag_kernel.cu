#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/user/kernels/diag_kernel.h"

namespace oneflow {
namespace {

template<typename T>
__global__ void vector_diagonal_kernel(T* out_buf, const T* in_buf, int32_t size, int32_t stride) {
  CUDA_1D_KERNEL_LOOP(i, size) { out_buf[i * stride] = in_buf[i]; }
}

template<typename T>
__global__ void matrix_diagonal_kernel(T* out_buf, const T* in_buf, int32_t size, int32_t stride) {
  CUDA_1D_KERNEL_LOOP(i, size) { out_buf[i] = in_buf[i * stride]; }
}

template<typename T>
struct DiagFunctor<DeviceType::kGPU, T> final {
  void operator()(DeviceCtx* ctx, T* out_buf, const T* in_buf, int32_t size, int32_t stride,
                  int32_t in_dim) {
    if (in_dim == 1) {
      vector_diagonal_kernel<<<BlocksNum4ThreadsNum(size * size), kCudaThreadsNumPerBlock, 0,
                               ctx->hip_stream()>>>(out_buf, in_buf, size, stride);
    } else {
      matrix_diagonal_kernel<<<BlocksNum4ThreadsNum(size * size), kCudaThreadsNumPerBlock, 0,
                               ctx->hip_stream()>>>(out_buf, in_buf, size, stride);
    }
  }
};

template<typename T>
struct DiagGradFunctor<DeviceType::kGPU, T> final {
  void operator()(DeviceCtx* ctx, T* dx_buf, const T* dy_buf, int32_t dx_cnt, int32_t dy_cnt,
                  int32_t stride, int32_t in_dim) {
    if (in_dim == 1) {
      matrix_diagonal_kernel<<<BlocksNum4ThreadsNum(dx_cnt), kCudaThreadsNumPerBlock, 0,
                               ctx->hip_stream()>>>(dx_buf, dy_buf, dx_cnt, stride);
    } else {
      vector_diagonal_kernel<<<BlocksNum4ThreadsNum(dy_cnt), kCudaThreadsNumPerBlock, 0,
                               ctx->hip_stream()>>>(dx_buf, dy_buf, dy_cnt, stride);
    }
  }
};

}  // namespace

REGISTER_DIAG_KERNELS(DeviceType::kGPU, half);
REGISTER_DIAG_KERNELS(DeviceType::kGPU, float);
REGISTER_DIAG_KERNELS(DeviceType::kGPU, double);
REGISTER_DIAG_KERNELS(DeviceType::kGPU, int8_t);
REGISTER_DIAG_KERNELS(DeviceType::kGPU, int32_t);
REGISTER_DIAG_KERNELS(DeviceType::kGPU, int64_t);

}  // namespace oneflow
