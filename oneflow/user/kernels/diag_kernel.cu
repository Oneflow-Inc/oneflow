#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/user/kernels/diag_kernel.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {
namespace {

template<typename T>
__global__ void vector_diagonal_kernel(T* out_buf, const T* in_buf, int32_t size, int32_t stride) {
  CUDA_1D_KERNEL_LOOP(i, size) { out_buf[i * stride] = in_buf[i]; }
}

template<typename T>
__global__ void matrix_diagonal_kernel(T* out_buf, const T* in_buf, int32_t size, int32_t stride) {
  CUDA_1D_KERNEL_LOOP(i, size) { out_buf[i] = in_buf[i * stride]; }
}

template<typename T>
struct DiagFunctor<DeviceType::kCUDA, T> final {
  void operator()(ep::Stream* stream, T* out_buf, const T* in_buf, int32_t size, int32_t stride,
                  int32_t in_dim) {
    if (in_dim == 1) {
      vector_diagonal_kernel<<<BlocksNum4ThreadsNum(size * size), kCudaThreadsNumPerBlock, 0,
                               stream->As<ep::CudaStream>()->hip_stream()>>>(out_buf, in_buf, size,
                                                                              stride);
    } else {
      matrix_diagonal_kernel<<<BlocksNum4ThreadsNum(size * size), kCudaThreadsNumPerBlock, 0,
                               stream->As<ep::CudaStream>()->hip_stream()>>>(out_buf, in_buf, size,
                                                                              stride);
    }
  }
};

template<typename T>
struct DiagGradFunctor<DeviceType::kCUDA, T> final {
  void operator()(ep::Stream* stream, T* dx_buf, const T* dy_buf, int32_t dx_cnt, int32_t dy_cnt,
                  int32_t stride, int32_t in_dim) {
    if (in_dim == 1) {
      matrix_diagonal_kernel<<<BlocksNum4ThreadsNum(dx_cnt), kCudaThreadsNumPerBlock, 0,
                               stream->As<ep::CudaStream>()->hip_stream()>>>(dx_buf, dy_buf,
                                                                              dx_cnt, stride);
    } else {
      vector_diagonal_kernel<<<BlocksNum4ThreadsNum(dy_cnt), kCudaThreadsNumPerBlock, 0,
                               stream->As<ep::CudaStream>()->hip_stream()>>>(dx_buf, dy_buf,
                                                                              dy_cnt, stride);
    }
  }
};

}  // namespace

REGISTER_DIAG_KERNELS(DeviceType::kCUDA, half);
REGISTER_DIAG_KERNELS(DeviceType::kCUDA, float);
REGISTER_DIAG_KERNELS(DeviceType::kCUDA, double);
REGISTER_DIAG_KERNELS(DeviceType::kCUDA, bool);
REGISTER_DIAG_KERNELS(DeviceType::kCUDA, uint8_t);
REGISTER_DIAG_KERNELS(DeviceType::kCUDA, int8_t);
REGISTER_DIAG_KERNELS(DeviceType::kCUDA, int32_t);
REGISTER_DIAG_KERNELS(DeviceType::kCUDA, int64_t);

}  // namespace oneflow
