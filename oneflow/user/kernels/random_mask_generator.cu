#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/random_mask_generator.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

constexpr int32_t kMinPackPerThread = 2;

using PackType = ulonglong2;

union Pack {
  PackType p_value;
  bool b_value[sizeof(PackType)];
};

__device__ bool GenMask(hiprandState* state, const float rate) {
  return hiprand_uniform(state) > rate;
}

__global__ void GenerateGpu(hiprandState* state, const int64_t n, const float rate, bool* mask) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState localState = state[id];
  PackType* pack_mask = reinterpret_cast<PackType*>(mask);
  Pack pack;
  CUDA_1D_KERNEL_LOOP(i, n / sizeof(PackType)) {
#pragma unroll
    for (int j = 0; j < sizeof(PackType); ++j) { pack.b_value[j] = GenMask(&localState, rate); }
    pack_mask[i] = pack.p_value;
  }
  const int32_t rem_cnt = n % sizeof(PackType);
  const int32_t rem_offset = n - rem_cnt;
  if (id < rem_cnt) { mask[id + rem_offset] = GenMask(&localState, rate); }
  state[id] = localState;
}

}  // namespace

void RandomMaskGenerator<DeviceType::kCUDA>::Generate(ep::Stream* stream, const int64_t n,
                                                      const float rate, bool* mask) {
  int32_t block_num = generator_->max_block_num();
  int32_t thread_num = generator_->max_thread_num();
  auto* curand_states = generator_->curand_states();
  const int32_t elem_cnt_per_block = thread_num * sizeof(PackType) * kMinPackPerThread;
  const int32_t block_num_final =
      std::min(static_cast<int32_t>((n + elem_cnt_per_block - 1) / elem_cnt_per_block), block_num);
  GenerateGpu<<<block_num_final, thread_num, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
      curand_states, n, rate, mask);
}

template class RandomMaskGenerator<DeviceType::kCUDA>;

}  // namespace oneflow
