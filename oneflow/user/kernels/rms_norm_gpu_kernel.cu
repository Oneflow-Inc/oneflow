#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/core/cuda/rms_norm.cuh"
#include <hipcub/hipcub.hpp>
#if CUDA_VERSION >= 11000
#include <hip/hip_bf16.h>
#endif  // CUDA_VERSION >= 11000

namespace oneflow {
namespace cuda {
namespace rms_norm {

template<typename SRC, typename DST, bool affine>
struct AffineStore {
  AffineStore(DST* y, int64_t row_size, const DST* weight)
      : y(y), weight(weight), row_size(row_size) {}

  template<int N>
  __device__ void store(const SRC* src, int64_t row, int64_t col) {
    layer_norm::Pack<DST, N> y_pack;
    layer_norm::Pack<DST, N> weight_pack;
    const int64_t offset = (row * row_size + col) / N;
    const int64_t weight_offset = col / N;
    if (affine) {
      weight_pack.storage =
          *(reinterpret_cast<const layer_norm::PackType<DST, N>*>(weight) + weight_offset);
    }
#pragma unroll
    for (int i = 0; i < N; ++i) {
      if (affine) {
        y_pack.elem[i] = static_cast<DST>(src[i]) * weight_pack.elem[i];
      } else {
        y_pack.elem[i] = static_cast<DST>(src[i]);
      }
    }
    *(reinterpret_cast<layer_norm::PackType<DST, N>*>(y) + offset) = y_pack.storage;
  }

  DST* y;
  const DST* weight;
  int64_t row_size;
};

// template<typename SRC, typename DST, bool do_scale>
// struct ScaleLoad {
//   ScaleLoad(const SRC* src, const SRC* gamma, int64_t row_size)
//       : src(src), gamma(gamma), row_size(row_size) {}
//   template<int N>
//   __device__ void load(DST* dst, int64_t row, int64_t col) const {
//     cuda::layer_norm::Pack<SRC, N> src_pack;
//     cuda::layer_norm::Pack<SRC, N> gamma_pack;
//     const int64_t offset = (row * row_size + col) / N;
//     const int64_t gamma_offset = col / N;
//     src_pack.storage = *(reinterpret_cast<const cuda::layer_norm::PackType<SRC, N>*>(src) +
//     offset); if (do_scale) {
//       gamma_pack.storage =
//           *(reinterpret_cast<const cuda::layer_norm::PackType<SRC, N>*>(gamma) + gamma_offset);
//     } else {
// #pragma unroll
//       for (int i = 0; i < N; ++i) { gamma_pack.elem[i] = static_cast<SRC>(1.f); }
//     }
// #pragma unroll
//     for (int i = 0; i < N; ++i) {
//       dst[i] = static_cast<DST>(src_pack.elem[i] * gamma_pack.elem[i]);
//     }
//   }
//   const SRC* src;
//   const SRC* gamma;
//   int64_t row_size;
// };

// template<typename SRC, typename DST, bool do_add>
// struct AddStore {
//   AddStore(const DST* add_to_output, DST* dst, int64_t row_size)
//       : add_to_output(add_to_output), dst(dst), row_size(row_size) {}
//   template<int N>
//   __device__ void store(const SRC* src, int64_t row, int64_t col) {
//     cuda::layer_norm::Pack<DST, N> add_to_output_pack;
//     cuda::layer_norm::Pack<DST, N> dst_pack;
//     const int64_t offset = (row * row_size + col) / N;
//     if (do_add) {
//       add_to_output_pack.storage =
//           *(reinterpret_cast<const cuda::layer_norm::PackType<DST, N>*>(add_to_output) + offset);
//     }
// #pragma unroll
//     for (int i = 0; i < N; ++i) {
//       if (do_add) {
//         dst_pack.elem[i] = static_cast<DST>(src[i]) + add_to_output_pack.elem[i];
//       } else {
//         dst_pack.elem[i] = static_cast<DST>(src[i]);
//       }
//     }
//     *(reinterpret_cast<cuda::layer_norm::PackType<DST, N>*>(dst) + offset) = dst_pack.storage;
//   }
//   const DST* add_to_output;
//   DST* dst;
//   int64_t row_size;
// };

// template<typename T>
// __inline__ __device__ T WarpReduce(T val) {
//   for (int mask = 16; mask > 0; mask /= 2) { val += __shfl_down_sync(0xffffffff, val, mask); }
//   return val;
// }

template<typename T, bool affine>
void RmsNormForwardGpu(ep::Stream* stream, const int64_t nrows, const int64_t ncols,
                       const double eps, const T* x_dptr, const T* w_dptr, T* y_dptr,
                       user_op::Tensor* inv_rms) {
  using ComputeType = typename layer_norm::DefaultComputeType<T>::type;
  layer_norm::DirectLoad<T, ComputeType> load(x_dptr, ncols);
  AffineStore<ComputeType, T, affine> store(y_dptr, ncols, w_dptr);
  DispatchRmsNorm<decltype(load), decltype(store), ComputeType>(
      stream->As<ep::CudaStream>()->hip_stream(), load, store, nrows, ncols, eps,
      inv_rms->mut_dptr<ComputeType>());
}

template<typename T>
void DispatchRmsNormForwardGpu(ep::Stream* stream, const int64_t nrows, const int64_t ncols,
                               const double eps, const T* x_dptr, const T* w_dptr, T* y_dptr,
                               user_op::Tensor* inv_rms) {
  if (w_dptr) {
    RmsNormForwardGpu<T, true>(stream, nrows, ncols, eps, x_dptr, w_dptr, y_dptr, inv_rms);
  } else {
    RmsNormForwardGpu<T, false>(stream, nrows, ncols, eps, x_dptr, w_dptr, y_dptr, inv_rms);
  }
}

// constexpr int tile_size = 32;
// constexpr int num_per_block = 4;
// constexpr int block_dim_x = 32;
// constexpr int block_dim_y = 32 / num_per_block;

// template<typename T, typename ComputeType>
// __global__ void LayerNormParamGrad(int rows, int cols, const T* __restrict__ dy,
//                                    const T* __restrict__ x, const ComputeType* __restrict__ mean,
//                                    const ComputeType* __restrict__ inv_var,
//                                    T* __restrict__ tmp_gamma_diff, T* __restrict__ tmp_beta_diff)
//                                    {
//   __shared__ ComputeType dgamma[32][33];
//   __shared__ ComputeType dbeta[32][33];
//   ComputeType dgamma_sum[num_per_block];
//   ComputeType dbeta_sum[num_per_block];
// #pragma unroll
//   for (int index = 0; index < num_per_block; ++index) {
//     dgamma_sum[index] = 0;
//     dbeta_sum[index] = 0;
//   }
//   const int col_id = blockIdx.x * blockDim.x + threadIdx.x;
//   if (col_id < cols) {
//     for (int i = blockIdx.y * tile_size + threadIdx.y; i < rows; i += tile_size * gridDim.y) {
// #pragma unroll
//       for (int index = 0; index < num_per_block; ++index) {
//         int row_id = i + index * blockDim.y;
//         if (row_id < rows) {
//           int offset = row_id * cols + col_id;
//           const ComputeType dy_val = static_cast<ComputeType>(dy[offset]);
//           const ComputeType x_val = static_cast<ComputeType>(x[offset]);
//           const ComputeType mean_val = mean[row_id];
//           const ComputeType inv_var_val = inv_var[row_id];
//           dgamma_sum[index] += dy_val * (x_val - mean_val) * inv_var_val;
//           dbeta_sum[index] += dy_val;
//         }
//       }
//     }
//   }
// #pragma unroll
//   for (int index = 0; index < num_per_block; ++index) {
//     dgamma[index * blockDim.y + threadIdx.y][threadIdx.x] = dgamma_sum[index];
//     dbeta[index * blockDim.y + threadIdx.y][threadIdx.x] = dbeta_sum[index];
//   }
//   __syncthreads();
// #pragma unroll
//   for (int index = 0; index < num_per_block; ++index) {
//     const int col_id = blockIdx.x * blockDim.x + threadIdx.y + index * blockDim.y;
//     if (col_id < cols) {
//       ComputeType gamma_sum = dgamma[threadIdx.x][threadIdx.y + index * blockDim.y];
//       ComputeType beta_sum = dbeta[threadIdx.x][threadIdx.y + index * blockDim.y];
//       ComputeType global_dgamma = WarpReduce<ComputeType>(gamma_sum);
//       ComputeType global_dbeta = WarpReduce<ComputeType>(beta_sum);
//       if (threadIdx.x == 0) {
//         const int offset = blockIdx.y * cols + col_id;
//         tmp_gamma_diff[offset] = global_dgamma;
//         tmp_beta_diff[offset] = global_dbeta;
//       }
//     }
//   }
// }

// template<typename T>
// int GetGirdDimY(const int64_t num_instances, const int64_t norm_size) {
//   using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
//   const int grid_dim_x = (norm_size + tile_size - 1) / tile_size;
//   const int max_grid_dim_y = (num_instances + tile_size - 1) / tile_size;
//   const int block_size = block_dim_x * block_dim_y;
//   int max_active_blocks = 0;
//   OF_CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(
//       &max_active_blocks, LayerNormParamGrad<T, ComputeType>, block_size, 0));
//   int waves = 1;
//   int dev;
//   OF_CUDA_CHECK(hipGetDevice(&dev));
//   int sm_count;
//   OF_CUDA_CHECK(hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev));
//   int num_blocks = max_active_blocks * sm_count * waves;
//   int grid_dim_y = std::min(max_grid_dim_y, static_cast<int>(num_blocks / grid_dim_x));
//   return std::max(grid_dim_y, 1);
// }

// template<typename T, bool do_scale, bool do_add>
// void LayerNormBackwardGpu(ep::Stream* stream, const int64_t num_instances, const int64_t
// norm_size,
//                           const T* dy_ptr, const T* x_ptr, const user_op::Tensor* mean,
//                           const user_op::Tensor* inv_variance, const T* gamma_ptr,
//                           const T* add_to_output_ptr, T* dx_ptr) {
//   using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
//   cuda::layer_norm::DirectLoad<T, ComputeType> load_x(x_ptr, norm_size);
//   ScaleLoad<T, ComputeType, do_scale> load_scaled_dy(dy_ptr, gamma_ptr, norm_size);
//   AddStore<ComputeType, T, do_add> store(add_to_output_ptr, dx_ptr, norm_size);
//   OF_CUDA_CHECK((cuda::layer_norm::DispatchLayerNormGrad<decltype(load_x),
//   decltype(load_scaled_dy),
//                                                          decltype(store), ComputeType>(
//       stream->As<ep::CudaStream>()->hip_stream(), load_x, load_scaled_dy, store,
//       mean->dptr<ComputeType>(), inv_variance->dptr<ComputeType>(), num_instances, norm_size)));
// }

// template<typename T, bool do_scale>
// void DispatchLayerNormBackwardDoAdd(ep::Stream* stream, const int64_t num_instances,
//                                     const int64_t norm_size, const T* dy_ptr, const T* x_ptr,
//                                     const user_op::Tensor* mean,
//                                     const user_op::Tensor* inv_variance, const T* gamma_ptr,
//                                     const T* add_to_output_ptr, T* dx_ptr) {
//   if (add_to_output_ptr != nullptr) {
//     LayerNormBackwardGpu<T, do_scale, true>(stream, num_instances, norm_size, dy_ptr, x_ptr,
//     mean,
//                                             inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
//   } else {
//     LayerNormBackwardGpu<T, do_scale, false>(stream, num_instances, norm_size, dy_ptr, x_ptr,
//     mean,
//                                              inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
//   }
// }

// template<typename T>
// void LaunchLayerNormBackward(ep::Stream* stream, const int64_t num_instances,
//                              const int64_t norm_size, const T* dy_ptr, const T* x_ptr,
//                              const user_op::Tensor* mean, const user_op::Tensor* inv_variance,
//                              const T* gamma_ptr, const T* add_to_output_ptr, T* dx_ptr) {
//   if (gamma_ptr != nullptr) {
//     DispatchLayerNormBackwardDoAdd<T, true>(stream, num_instances, norm_size, dy_ptr, x_ptr,
//     mean,
//                                             inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
//   } else {
//     DispatchLayerNormBackwardDoAdd<T, false>(stream, num_instances, norm_size, dy_ptr, x_ptr,
//     mean,
//                                              inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
//   }
// }

}  // namespace rms_norm

template<typename T>
class RmsNormGpuKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  RmsNormGpuKernel() = default;
  ~RmsNormGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    user_op::Tensor* inv_rms = ctx->Tensor4ArgNameAndIndex("inv_rms", 0);
    const double eps = ctx->Attr<double>("epsilon");
    const Shape& normalized_shape = ctx->Attr<Shape>("normalized_shape");
    const int64_t ncols = normalized_shape.elem_cnt();
    const int64_t nrows = inv_rms->shape_view().elem_cnt();
    CHECK_EQ(x->shape_view().elem_cnt(), ncols * nrows);

    const T* weight_dptr = nullptr;
    if (ctx->has_input("weight", 0)) {
      const auto* weight = ctx->Tensor4ArgNameAndIndex("weight", 0);
      CHECK_EQ(weight->shape_view().elem_cnt(), ncols);
      weight_dptr = weight->dptr<T>();
    }
    rms_norm::DispatchRmsNormForwardGpu<T>(ctx->stream(), nrows, ncols, eps, x->dptr<T>(),
                                           weight_dptr, y->mut_dptr<T>(), inv_rms);
  };
};

#define REGISTER_RMS_NORM_CUDA_KERNEL(dtype)                           \
  REGISTER_USER_KERNEL("rms_norm")                                     \
      .SetCreateFn<RmsNormGpuKernel<dtype>>()                          \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value));

REGISTER_RMS_NORM_CUDA_KERNEL(float)
REGISTER_RMS_NORM_CUDA_KERNEL(double)
REGISTER_RMS_NORM_CUDA_KERNEL(half)
#if CUDA_VERSION >= 11000
REGISTER_RMS_NORM_CUDA_KERNEL(hip_bfloat16)
#endif

// template<typename T>
// class LayerNormGradGpuKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
//  public:
//   LayerNormGradGpuKernel() = default;
//   ~LayerNormGradGpuKernel() = default;

//  private:
//   using user_op::OpKernel::Compute;
//   bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
//   void Compute(user_op::KernelComputeContext* ctx) const override {
//     const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
//     const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
//     const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
//     const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
//     user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
//     const int64_t num_instances = mean->shape_view().elem_cnt();
//     const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
//     const T* gamma_ptr = nullptr;
//     if (ctx->has_input("gamma", 0)) {
//       gamma_ptr = ctx->Tensor4ArgNameAndIndex("gamma", 0)->dptr<T>();
//     }
//     const T* add_to_output_ptr = nullptr;
//     if (ctx->has_input("_add_to_output", 0)) {
//       const user_op::Tensor* add_to_output = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
//       CHECK_EQ(add_to_output->data_type(), dx->data_type());
//       CHECK_EQ(add_to_output->shape_view(), dx->shape_view());
//       add_to_output_ptr = add_to_output->dptr<T>();
//     }
//     LaunchLayerNormBackward<T>(ctx->stream(), num_instances, norm_size, dy->dptr<T>(),
//     x->dptr<T>(),
//                                mean, inv_variance, gamma_ptr, add_to_output_ptr,
//                                dx->mut_dptr<T>());
//   };
// };

// #define REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(dtype)                                        \
//   REGISTER_USER_KERNEL("layer_norm_grad")                                                  \
//       .SetCreateFn<LayerNormGradGpuKernel<dtype>>()                                        \
//       .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                     \
//                        && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value))    \
//       .SetInplaceProposalFn(                                                               \
//           [](const user_op::InferContext& ctx,                                             \
//              const user_op::AddInplaceArgPair& AddInplaceArgPairFn) -> Maybe<void> {       \
//             if (ctx.has_input("_add_to_output", 0)) {                                      \
//               OF_RETURN_IF_ERROR(AddInplaceArgPairFn("dx", 0, "_add_to_output", 0, true)); \
//             }                                                                              \
//             return Maybe<void>::Ok();                                                      \
//           });

// REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(float)
// REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(double)
// REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(half)
// #if CUDA_VERSION >= 11000
// REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(hip_bfloat16)
// #endif

// template<typename T>
// class LayerNormParamGradGpuKernel final : public user_op::OpKernel,
//                                           public user_op::CudaGraphSupport {
//  public:
//   LayerNormParamGradGpuKernel() = default;
//   ~LayerNormParamGradGpuKernel() = default;

//  private:
//   using user_op::OpKernel::Compute;
//   bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
//   void Compute(user_op::KernelComputeContext* ctx) const override {
//     const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
//     const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
//     const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
//     const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
//     const int64_t num_instances = mean->shape_view().elem_cnt();
//     const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
//     user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
//     const DataType data_type = dy->data_type();
//     const int grid_dim_x = (norm_size + tile_size - 1) / tile_size;
//     const int grid_dim_y = GetGirdDimY<T>(num_instances, norm_size);
//     const size_t tmp_gamma_diff_size = grid_dim_y * norm_size * sizeof(T);
//     T* tmp_gamma_diff_ptr = reinterpret_cast<T*>(tmp_buffer->mut_dptr());
//     T* tmp_beta_diff_ptr = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() +
//     tmp_gamma_diff_size); T* reduce_buf_ptr =
//         reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + 2 * tmp_gamma_diff_size);
//     using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
//     LayerNormParamGrad<T, ComputeType><<<dim3(grid_dim_x, grid_dim_y), dim3(32, 32 /
//     num_per_block),
//                                          0,
//                                          ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
//         num_instances, norm_size, dy->dptr<T>(), x->dptr<T>(), mean->dptr<ComputeType>(),
//         inv_variance->dptr<ComputeType>(), tmp_gamma_diff_ptr, tmp_beta_diff_ptr);
//     const int32_t m = norm_size;
//     const int32_t n = 1;
//     const int32_t k = grid_dim_y;
//     std::unique_ptr<ep::primitive::Fill> fill =
//         ep::primitive::NewPrimitive<ep::primitive::FillFactory>(ctx->stream()->device_type(),
//                                                                 data_type);
//     CHECK(fill);
//     fill->Launch(ctx->stream(), reduce_buf_ptr, 1.0, grid_dim_y);
//     std::unique_ptr<ep::primitive::Matmul> matmul =
//         ep::primitive::NewPrimitive<ep::primitive::MatmulFactory>(
//             ctx->stream()->device_type(), data_type, ep::primitive::BlasTransposeType::T,
//             ep::primitive::BlasTransposeType::N);
//     CHECK(matmul);
//     if (ctx->has_output("gamma_diff", 0)) {
//       user_op::Tensor* gamma_diff = ctx->Tensor4ArgNameAndIndex("gamma_diff", 0);
//       matmul->Launch(ctx->stream(), m, n, k, 1.0, tmp_gamma_diff_ptr, reduce_buf_ptr, 0.0,
//                      gamma_diff->mut_dptr());
//     }
//     if (ctx->has_output("beta_diff", 0)) {
//       user_op::Tensor* beta_diff = ctx->Tensor4ArgNameAndIndex("beta_diff", 0);
//       matmul->Launch(ctx->stream(), m, n, k, 1.0, tmp_beta_diff_ptr, reduce_buf_ptr, 0.0,
//                      beta_diff->mut_dptr());
//     }
//   };
// };

// #define REGISTER_LAYER_NORM_PARAM_GRAD_GPU_KERNEL(dtype)                                    \
//   REGISTER_USER_KERNEL("layer_norm_param_grad")                                             \
//       .SetCreateFn<LayerNormParamGradGpuKernel<dtype>>()                                    \
//       .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                      \
//                        && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value))     \
//       .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                   \
//         const int64_t begin_params_axis = ctx->Attr<int64_t>("begin_params_axis");          \
//         const bool has_gamma_diff = ctx->has_output("gamma_diff", 0);                       \
//         const bool has_beta_diff = ctx->has_output("beta_diff", 0);                         \
//         const auto& dy = ctx->InputTensorDesc("dy", 0);                                     \
//         const int64_t num_instances = dy.shape().Count(0, begin_params_axis);               \
//         const int64_t norm_size = dy.shape().Count(begin_params_axis);                      \
//         const int grid_dim_y = GetGirdDimY<dtype>(num_instances, norm_size);                \
//         size_t tmp_buffer_size = (2 * grid_dim_y * norm_size + grid_dim_y) * sizeof(dtype); \
//         return tmp_buffer_size;                                                             \
//       });

// REGISTER_LAYER_NORM_PARAM_GRAD_GPU_KERNEL(float)
// REGISTER_LAYER_NORM_PARAM_GRAD_GPU_KERNEL(double)
// REGISTER_LAYER_NORM_PARAM_GRAD_GPU_KERNEL(half)
// #if CUDA_VERSION >= 11000
// REGISTER_LAYER_NORM_PARAM_GRAD_GPU_KERNEL(hip_bfloat16)
// #endif

}  // namespace cuda
}  // namespace oneflow
