/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <cstdint>
#ifdef WITH_CUDA
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/dim_gather_kernel_util.h"

namespace oneflow {

namespace user_op {

template<typename IN_T, typename IDX_T>
__global__ void DoCUDADimGather(const DimOpIndexNdHelper<IDX_T> input_nd_helper,
                                const DimOpIndexNdHelper<IDX_T> index_nd_helper, int ndim,
                                int64_t elem_cnt, int32_t dim, const IDX_T* index,
                                const IN_T* input, IN_T* output) {
  DoDimGather<IN_T, IDX_T>(input_nd_helper, index_nd_helper, ndim, elem_cnt, dim, index, input,
                           output);
}

template<typename IDX_T, typename IN_T>
struct DimGatherFunctor<DeviceType::kGPU, IN_T, IDX_T> final {
  void operator()(DeviceCtx* ctx, const DimOpIndexNdHelper<IDX_T>& input_nd_helper,
                  const DimOpIndexNdHelper<IDX_T>& index_nd_helper, int ndim, int64_t elem_cnt,
                  int32_t dim, const IDX_T* index, const IN_T* input, IN_T* output) {
    RUN_CUDA_KERNEL((DoCUDADimGather<IN_T, IDX_T>), ctx, BlocksNum4ThreadsNum(elem_cnt),
                    input_nd_helper, index_nd_helper, ndim, elem_cnt, dim, index, input, output);
  }
};

// float16 special case of DimGatherFunctor template
template<typename IDX_T>
struct DimGatherFunctor<DeviceType::kGPU, float16, IDX_T> final {
  void operator()(DeviceCtx* ctx, const DimOpIndexNdHelper<IDX_T>& input_nd_helper,
                  const DimOpIndexNdHelper<IDX_T>& index_nd_helper, int ndim, int64_t elem_cnt,
                  int32_t dim, const IDX_T* index, const float16* input, float16* output) {
    RUN_CUDA_KERNEL((DoCUDADimGather<half, IDX_T>), ctx, BlocksNum4ThreadsNum(elem_cnt),
                    input_nd_helper, index_nd_helper, ndim, elem_cnt, dim, index,
                    reinterpret_cast<const half*>(input), reinterpret_cast<half*>(output));
  }
};

template<typename IN_T, typename IDX_T>
__global__ void DoCUDAScatterDimAdd(const DimOpIndexNdHelper<IDX_T> input_nd_helper,
                                    const DimOpIndexNdHelper<IDX_T> output_nd_helper, int ndim,
                                    int64_t elem_cnt, int32_t dim, const IDX_T* index,
                                    const IN_T* input, IN_T* output) {
  DoDimScatterAdd<IN_T, IDX_T>(input_nd_helper, output_nd_helper, ndim, elem_cnt, dim, index, input,
                               output);
}

template<typename IN_T, typename IDX_T>
struct DimScatterAddFunctor<DeviceType::kGPU, IN_T, IDX_T> final {
  void operator()(DeviceCtx* ctx, const DimOpIndexNdHelper<IDX_T>& input_nd_helper,
                  const DimOpIndexNdHelper<IDX_T>& output_nd_helper, int ndim, int64_t elem_cnt,
                  int32_t dim, const IDX_T* index, const IN_T* input, IN_T* output) {
    RUN_CUDA_KERNEL((DoCUDAScatterDimAdd<IN_T, IDX_T>), ctx, BlocksNum4ThreadsNum(elem_cnt),
                    input_nd_helper, output_nd_helper, ndim, elem_cnt, dim, index, input, output);
  }
};

// float16 special case of DimScatterAddFunctor template
template<typename IDX_T>
struct DimScatterAddFunctor<DeviceType::kGPU, float16, IDX_T> final {
  void operator()(DeviceCtx* ctx, const DimOpIndexNdHelper<IDX_T>& input_nd_helper,
                  const DimOpIndexNdHelper<IDX_T>& output_nd_helper, int ndim, int64_t elem_cnt,
                  int32_t dim, const IDX_T* index, const float16* input, float16* output) {
    RUN_CUDA_KERNEL((DoCUDAScatterDimAdd<half, IDX_T>), ctx, BlocksNum4ThreadsNum(elem_cnt),
                    input_nd_helper, output_nd_helper, ndim, elem_cnt, dim, index,
                    reinterpret_cast<const half*>(input), reinterpret_cast<half*>(output));
  }
};

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_DIM_GATHER_FUNCTOR, (DeviceType::kGPU),
                                 DIM_GATHER_SCATTER_DATA_TYPE_GPU_SEQ, INDEX_DATA_TYPE_SEQ);
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_DIM_SCATTER_ADD_FUNCTOR, (DeviceType::kGPU),
                                 DIM_GATHER_SCATTER_DATA_TYPE_GPU_SEQ, INDEX_DATA_TYPE_SEQ);

}  // namespace user_op
}  // namespace oneflow

#endif  // WITH_CUDA
