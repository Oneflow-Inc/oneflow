#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/in_top_k_kernel_util.h"

namespace oneflow {

namespace {

template<typename IDX>
__global__ void InTopkGpu(const int targets_num, const int classes_num, const IDX* targets,
                          const float* predictions, const int k, int8_t* out) {
  CUDA_1D_KERNEL_LOOP(idx, targets_num) {
    IDX target = targets[idx];
    bool cannot_say = (target >= classes_num) || !isfinite(predictions[idx * classes_num + target]);

    int32_t more_probable_classes = 0;
    if (!cannot_say) {
      const float target_prediction = predictions[idx * classes_num + target];
      FOR_RANGE(int32_t, class_idx, 0, classes_num) {
        float pred = predictions[idx * classes_num + class_idx];

        if (!isfinite(pred)) {
          cannot_say = true;
          break;
        } else if (pred > target_prediction) {
          ++more_probable_classes;
          if (more_probable_classes > k) break;
        }
      }
    }
    out[idx] = cannot_say ? false : (more_probable_classes < k);
  }
}

}  // namespace

template<typename IDX>
struct InTopkKernelUtil<DeviceType::kGPU, IDX> {
  static void InTopk(DeviceCtx* ctx, const int targets_num, const int classes_num,
                     const IDX* targets, const float* predictions, const int k, int8_t* out) {
    RUN_CUDA_KERNEL((InTopkGpu<IDX>), ctx, targets_num, targets_num, classes_num, targets,
                    predictions, k, out);
  }
};

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_IN_TOP_K_FUNCTOR, (DeviceType::kGPU),
                                 INDEX_DATA_TYPE_SEQ)

}  // namespace oneflow
