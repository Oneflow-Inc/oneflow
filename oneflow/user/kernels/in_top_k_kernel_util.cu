#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/in_top_k_kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void InTopkGpu(const int instance_num, const int classes_num, const T* targets,
                          const float* predictions, const int k, int8_t* out) {
  CUDA_1D_KERNEL_LOOP(idx, instance_num) {
    T target = targets[idx];
    bool cannot_say = (target >= classes_num) || !isfinite(predictions[idx * classes_num + target]);

    int32_t more_probable_classes = 0;
    if (!cannot_say) {
      const float target_prediction = predictions[idx * classes_num + target];
      FOR_RANGE(int32_t, class_idx, 0, classes_num) {
        float pred = predictions[idx * classes_num + class_idx];

        if (!isfinite(pred)) {
          cannot_say = true;
          break;
        } else if (pred > target_prediction) {
          ++more_probable_classes;
          if (more_probable_classes > k) break;
        }
      }
    }
    out[idx] = cannot_say ? false : (more_probable_classes < k);
  }
}

}  // namespace

template<typename T>
struct InTopkKernelUtil<DeviceType::kGPU, T> {
  static void InTopk(DeviceCtx* ctx, const int instance_num, const int classes_num,
                     const T* targets, const float* predictions, const int k, int8_t* out) {
    RUN_CUDA_KERNEL((InTopkGpu<T>), ctx, instance_num, instance_num, classes_num, targets,
                    predictions, k, out);
  }
};

#define INSTANTIATE_IN_TOP_K_KERNEL_UTIL_GPU(cpp_data_type, data_type) \
  template struct InTopkKernelUtil<DeviceType::kGPU, cpp_data_type>;

OF_PP_FOR_EACH_TUPLE(INSTANTIATE_IN_TOP_K_KERNEL_UTIL_GPU, INDEX_DATA_TYPE_SEQ)

#undef INSTANTIATE_IN_TOP_K_KERNEL_UTIL_GPU

}  // namespace oneflow
