#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/user/kernels/math_unary_elementwise_func.h"

namespace oneflow {

namespace {

template<typename T>
struct PxyForwardFunctor {
  __device__ T Compute(T pxy) const {
    const T pxy_sigmoid =
        static_cast<T>(1.0)
        / (static_cast<T>(1.0) + ExpFunctor<T>::Forward(static_cast<T>(-1.0) * pxy));
    return pxy_sigmoid * static_cast<T>(2.0) - static_cast<T>(0.5);
  }
};

template<typename T>
struct PwhForwardFunctor {
  __device__ T Compute(T pwh, T anchors) const {
    const T pwh_sigmoid =
        static_cast<T>(1.0)
        / (static_cast<T>(1.0) + ExpFunctor<T>::Forward(static_cast<T>(-1.0) * pwh));
    return static_cast<T>(4.0) * pwh_sigmoid * pwh_sigmoid * anchors;
  }
};

template<typename T>
struct PxyBackwardFunctor {
  __device__ T Compute(T pxy) const {
    const T minus_pxy_exp = ExpFunctor<T>::Forward(static_cast<T>(-1.0) * pxy);
    const T minus_pxy_exp_1 = static_cast<T>(1.0) + minus_pxy_exp;
    return static_cast<T>(2.0) * minus_pxy_exp / (minus_pxy_exp_1 * minus_pxy_exp_1);
  }
};

template<typename T>
struct PwhBackwardFunctor {
  __device__ T Compute(T minus_pwh_exp, T minus_pwh_exp_1, T anchors) const {
    return static_cast<T>(8.0) * anchors * minus_pwh_exp / pow(minus_pwh_exp_1, 3);
  }
};

template<>
struct PwhBackwardFunctor<half> {
  __device__ half Compute(half minus_pwh_exp, half minus_pwh_exp_1, half anchors) const {
    return static_cast<half>(8.0) * anchors * minus_pwh_exp
           / (minus_pwh_exp_1 * minus_pwh_exp_1 * minus_pwh_exp_1);
  }
};

template<typename T>
struct AnchorsBackwardFunctor {
  __device__ T Compute(T minus_pwh_exp_1, T anchors) const {
    return static_cast<T>(4.0) / (minus_pwh_exp_1 * minus_pwh_exp_1);
  }
};

template<typename FUNCTOR_PXY, typename FUNCTOR_PWH, typename T>
__global__ void FusedGetPboxForward(FUNCTOR_PXY pxy_functor, FUNCTOR_PWH pwh_functor, const int n,
                                    const T* pxy, const T* pwh, const T* anchors, T* pbox,
                                    const int64_t cols) {
  CUDA_1D_KERNEL_LOOP_T(int64_t, i, n) {
    const int64_t extra_cols = i - (i % cols);
    pbox[i + extra_cols] = pxy_functor.Compute(pxy[i]);
    pbox[i + cols + extra_cols] = pwh_functor.Compute(pwh[i], anchors[i]);
  }
}

template<typename FUNCTOR_PXY, typename FUNCTOR_PWH, typename FUNCTOR_ANCHORS, typename T>
__global__ void FusedGetPboxBackward(FUNCTOR_PXY pxy_functor, FUNCTOR_PWH pwh_functor,
                                     FUNCTOR_ANCHORS anchors_functor, const int n, const T* pxy,
                                     const T* pwh, const T* anchors, const T* pbox_diff,
                                     T* pxy_diff, T* pwh_diff, T* anchors_diff) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T minus_pwh_exp = ExpFunctor<T>::Forward(static_cast<T>(-1.0) * pwh[i]);
    const T minus_pwh_exp_1 = minus_pwh_exp + static_cast<T>(1.0);
    pxy_diff[i] = pxy_functor.Compute(pxy[i]) * pbox_diff[i];

    const T anchors_i = anchors[i];
    const T pbox_diff_i_n = pbox_diff[i + n];
    pwh_diff[i] = pwh_functor.Compute(minus_pwh_exp, minus_pwh_exp_1, anchors_i) * pbox_diff_i_n;
    anchors_diff[i] = anchors_functor.Compute(minus_pwh_exp_1, anchors_i) * pbox_diff_i_n;
  }
}

}  // namespace

template<typename T>
class FusedGetPboxKernel final : public user_op::OpKernel {
 public:
  FusedGetPboxKernel() = default;
  ~FusedGetPboxKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* pxy = ctx->Tensor4ArgNameAndIndex("pxy", 0);
    const user_op::Tensor* pwh = ctx->Tensor4ArgNameAndIndex("pwh", 0);
    const user_op::Tensor* anchors = ctx->Tensor4ArgNameAndIndex("anchors", 0);

    user_op::Tensor* pbox = ctx->Tensor4ArgNameAndIndex("pbox", 0);
    const ShapeView& pxy_shape = pxy->shape_view();
    const int64_t elem_cnt = pxy_shape.elem_cnt();
    const int64_t cols = pxy_shape.At(1);

    PxyForwardFunctor<T> pxy_functor{};
    PwhForwardFunctor<T> pwh_functor{};
    RUN_CUDA_KERNEL((FusedGetPboxForward<decltype(pxy_functor), decltype(pwh_functor), T>),
                    ctx->stream(), elem_cnt, pxy_functor, pwh_functor, elem_cnt, pxy->dptr<T>(),
                    pwh->dptr<T>(), anchors->dptr<T>(), pbox->mut_dptr<T>(), cols);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_GET_PBOX_KERNEL(dtype)   \
  REGISTER_USER_KERNEL("fused_get_pbox")        \
      .SetCreateFn<FusedGetPboxKernel<dtype>>() \
      .SetIsMatchedHob((user_op::HobDataType("pbox", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_GET_PBOX_KERNEL(float)
REGISTER_FUSED_GET_PBOX_KERNEL(double)
REGISTER_FUSED_GET_PBOX_KERNEL(half)

template<typename T>
class FusedGetPboxGradKernel final : public user_op::OpKernel {
 public:
  FusedGetPboxGradKernel() = default;
  ~FusedGetPboxGradKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* pxy = ctx->Tensor4ArgNameAndIndex("pxy", 0);
    const user_op::Tensor* pwh = ctx->Tensor4ArgNameAndIndex("pwh", 0);
    const user_op::Tensor* anchors = ctx->Tensor4ArgNameAndIndex("anchors", 0);
    const user_op::Tensor* pbox_diff = ctx->Tensor4ArgNameAndIndex("pbox_diff", 0);

    user_op::Tensor* pxy_diff = ctx->Tensor4ArgNameAndIndex("pxy_diff", 0);
    user_op::Tensor* pwh_diff = ctx->Tensor4ArgNameAndIndex("pwh_diff", 0);
    user_op::Tensor* anchors_diff = ctx->Tensor4ArgNameAndIndex("anchors_diff", 0);

    PxyBackwardFunctor<T> pxy_functor{};
    PwhBackwardFunctor<T> pwh_functor{};
    AnchorsBackwardFunctor<T> anchors_functor{};
    const int64_t elem_cnt = pxy->shape_view().elem_cnt();

    RUN_CUDA_KERNEL((FusedGetPboxBackward<decltype(pxy_functor), decltype(pwh_functor),
                                          decltype(anchors_functor), T>),
                    ctx->stream(), elem_cnt, pxy_functor, pwh_functor, anchors_functor, elem_cnt,
                    pxy->dptr<T>(), pwh->dptr<T>(), anchors->dptr<T>(), pbox_diff->dptr<T>(),
                    pxy_diff->mut_dptr<T>(), pwh_diff->mut_dptr<T>(), anchors_diff->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_GET_PBOX_GRAD_KERNEL(dtype)  \
  REGISTER_USER_KERNEL("fused_get_pbox_grad")       \
      .SetCreateFn<FusedGetPboxGradKernel<dtype>>() \
      .SetIsMatchedHob((user_op::HobDataType("pxy_diff", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_GET_PBOX_GRAD_KERNEL(float)
REGISTER_FUSED_GET_PBOX_GRAD_KERNEL(double)
REGISTER_FUSED_GET_PBOX_GRAD_KERNEL(half)

}  // namespace oneflow
