/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <hipcub/hipcub.hpp>
#include <>
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/user/kernels/radix_sort.cuh"

namespace oneflow {


template<typename T>
class CudaModeKernel final : public user_op::OpKernel {
 public:
  CudaModeKernel() = default;
  ~CudaModeKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
   
   
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};


#define REGISTER_CUDA_MODE_KERNEL(dtype)                                                   \
  REGISTER_USER_KERNEL("mode")                                                             \
      .SetCreateFn<CudaModeKernel<dtype>>()                                                \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                     \
                       && (user_op::HobDataType("input", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) -> size_t {                        \
        const Shape& in_shape = ctx->InputShape("input", 0);                               \
        const int64_t instance_size = in_shape.dim_vec().back();                           \
        const int64_t instance_num = in_shape.elem_cnt() / instance_size;                  \
        return 0;                                                                          \
      });

REGISTER_CUDA_MODE_KERNEL(float)
REGISTER_CUDA_MODE_KERNEL(double)
REGISTER_CUDA_MODE_KERNEL(int8_t)
REGISTER_CUDA_MODE_KERNEL(uint8_t)
REGISTER_CUDA_MODE_KERNEL(int32_t)
REGISTER_CUDA_MODE_KERNEL(int64_t)

}  // namespace oneflow
