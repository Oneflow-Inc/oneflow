#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/ep/include/primitive/memcpy.h"
#include "oneflow/core/common/str_util.h"
#include "oneflow/core/ep/include/primitive/copy_nd.h"
#include "oneflow/core/ep/include/primitive/permute.h"
#include "oneflow/core/ep/include/primitive/add.h"

namespace oneflow {

namespace {

void DumpToFile(ep::Stream* stream, std::string filename, int64_t parallel_id, size_t data_size,
                const void* ptr) {
  void* host_ptr;
  OF_CUDA_CHECK(hipHostMalloc(&host_ptr, data_size));
  std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
      ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                ep::primitive::MemcpyKind::kDtoH);
  CHECK(copyd2h_primitive);
  copyd2h_primitive->Launch(stream, host_ptr, ptr, data_size);
  CHECK_JUST(stream->Sync());
  std::ofstream dx_os;
  dx_os.open(StrCat("test/" + filename + "_", parallel_id));
  dx_os.write(reinterpret_cast<char*>(host_ptr), data_size);
  dx_os.close();
  OF_CUDA_CHECK(hipHostFree(host_ptr));
}

}  // namespace

template<typename T>
void ConcatKernel(ep::Stream* stream, DataType data_type, const int64_t rows,
                  const int64_t out_cols, const std::vector<int64_t>& in_cols,
                  const std::vector<const void*>& in_ptrs, void* out_ptr) {
  auto primitive = ep::primitive::NewPrimitive<ep::primitive::CopyNdFactory>(DeviceType::kCUDA, 2);
  int64_t out_col_offset = 0;
  for (int i = 0; i < in_cols.size(); ++i) {
    if (in_cols.at(i) > 0) {
      DimVector dst_shape = {rows, out_cols};
      DimVector dst_pos_vec = {0, out_col_offset};
      DimVector src_shape = {rows, in_cols.at(i)};
      DimVector src_pos_vec = {0, 0};
      DimVector extent_vec = {rows, in_cols.at(i)};
      primitive->Launch(stream, data_type, 2, out_ptr, dst_shape.data(), dst_pos_vec.data(),
                        in_ptrs.at(i), src_shape.data(), src_pos_vec.data(), extent_vec.data());
    }
    out_col_offset += in_cols.at(i);
  }
}

template<typename T>
void BatchMatmul(ep::Stream* stream, DataType data_type, const bool transpose_b,
                 const int64_t batch_size, const int64_t m, const int64_t n, const int64_t k,
                 const T* in_a, const T* in_b, T* out) {
  float alpha = 1.0f;
  float beta = 0.0f;
  int lda = k;
  int ldb;
  int ldc = n;
  int stride_a = m * k;
  int stride_b = k * n;
  int stride_c = m * n;
  hipblasOperation_t trans_b{};
  if (transpose_b) {
    trans_b = HIPBLAS_OP_T;
    ldb = k;
  } else {
    trans_b = HIPBLAS_OP_N;
    ldb = n;
  }
#if CUDA_VERSION >= 11000
  hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;
#else
  hipblasGemmAlgo_t algo =
      (data_type == DataType::kFloat16) ? CUBLAS_GEMM_DFALT_TENSOR_OP : HIPBLAS_GEMM_DEFAULT;
#endif
  hipDataType cuda_data_type;
  hipDataType compute_type = HIP_R_32F;
  if (data_type == DataType::kFloat16) {
    cuda_data_type = HIP_R_16F;
  } else if (data_type == DataType::kFloat) {
    cuda_data_type = HIP_R_32F;
  } else {
    UNIMPLEMENTED();
  }
  OF_CUBLAS_CHECK(hipblasGemmStridedBatchedEx(
      stream->As<ep::CudaStream>()->cublas_handle(), trans_b, HIPBLAS_OP_N, n, m, k, &alpha, in_b,
      cuda_data_type, ldb, stride_b, in_a, cuda_data_type, lda, stride_a, &beta, out,
      cuda_data_type, ldc, stride_c, batch_size, compute_type, algo));
}

__global__ void GenerateTrilIndicesGpu(const int32_t concat_dim, const int32_t pad_dim,
                                       int32_t* tril_indices) {
  for (int row = threadIdx.y; row < concat_dim; row += blockDim.y) {
    for (int col = threadIdx.x; col < concat_dim; col += blockDim.x) {
      if (col > row) {
        int in_index = row * pad_dim + col;
        int idx = (col * (col - 1) / 2) + row;
        tril_indices[idx] = in_index;
      }
    }
  }
}

template<typename T>
__global__ void GatherConcatGpu(int64_t batch_size, int64_t out_dim, int64_t tril_dim,
                                int64_t pad_dim, int64_t embedding_size,
                                const int32_t* tril_indices, const T* matmul_out,
                                const T* dense_feature_ptr, T* out_ptr) {
  for (int row = blockIdx.x; row < batch_size; row += gridDim.x) {
    const T* row_matmul = matmul_out + row * pad_dim * pad_dim;
    const T* row_dense_feature = dense_feature_ptr + row * embedding_size;
    T* row_out = out_ptr + row * out_dim;
    for (int col = threadIdx.x; col < out_dim; col += blockDim.x) {
      T out_val = 0;
      if (col < embedding_size) {
        out_val = row_dense_feature[col];
      } else if (col < embedding_size + tril_dim) {
        int32_t index = tril_indices[col - embedding_size];
        out_val = row_matmul[index];
      }
      row_out[col] = out_val;
    }
  }
}

template<typename T>
__global__ void ScatterSplitGpu(int64_t batch_size, int64_t out_dim, int64_t pad_dim,
                                int64_t concat_dim, int64_t embedding_size, const T* dy,
                                T* dense_feature_grad, T* matmul_out_grad) {
  for (int row = blockIdx.x; row < batch_size; row += gridDim.x) {
    const T* row_dy = dy + row * out_dim;
    T* row_dense_feature_grad = dense_feature_grad + row * embedding_size;
    T* row_matmul_out_grad = matmul_out_grad + row * pad_dim * pad_dim;
    for (int col = threadIdx.x; col < embedding_size + pad_dim * pad_dim; col += blockDim.x) {
      if (col < embedding_size) {
        row_dense_feature_grad[col] = row_dy[col];
      } else {
        int sparse_col_id = col - embedding_size;
        int i = sparse_col_id / pad_dim;
        int j = sparse_col_id - i * pad_dim;
        T sparse_grad = 0;
        if (j > i && j < concat_dim) {
          int dy_idx = (j * (j - 1) / 2) + i;
          sparse_grad = row_dy[embedding_size + dy_idx];
        }
        row_matmul_out_grad[sparse_col_id] = sparse_grad;
      }
    }
  }
}

constexpr int64_t kFusedInteractionBlockSize = 256;

int GetNumBlocks(const int num_instances) {
  return std::min(static_cast<int>(num_instances), kCudaMaxBlocksNum);
}

void GenerateTrilIndices(ep::Stream* stream, const int32_t concat_dim, const int32_t pad_dim,
                         int32_t* tril_indices) {
  GenerateTrilIndicesGpu<<<1, dim3(32, 32), 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
      concat_dim, pad_dim, tril_indices);
}

// out_dim 480 tril_dim 351 pad_dim 32 concat_dim 27
template<typename T>
void GatherConcatKernel(ep::Stream* stream, int64_t batch_size, int64_t out_dim, int64_t tril_dim,
                        int64_t pad_dim, int64_t concat_dim, int64_t embedding_size,
                        const int32_t* tril_indices, const T* matmul_out,
                        const T* dense_feature_ptr, T* out_ptr) {
  GatherConcatGpu<<<GetNumBlocks(batch_size), kFusedInteractionBlockSize, 0,
                    stream->As<ep::CudaStream>()->hip_stream()>>>(
      batch_size, out_dim, tril_dim, pad_dim, embedding_size, tril_indices, matmul_out,
      dense_feature_ptr, out_ptr);
}

template<typename T>
void ScatterSplitKernel(ep::Stream* stream, int64_t batch_size, int64_t out_dim, int64_t pad_dim,
                        int64_t concat_dim, int64_t embedding_size, const T* dy,
                        T* dense_feature_grad, T* matmul_out_grad_ptr) {
  ScatterSplitGpu<<<GetNumBlocks(batch_size), kFusedInteractionBlockSize, 0,
                    stream->As<ep::CudaStream>()->hip_stream()>>>(
      batch_size, out_dim, pad_dim, concat_dim, embedding_size, dy, dense_feature_grad,
      matmul_out_grad_ptr);
}

template<typename T>
__global__ void SliceAddGpu(const int64_t batch_size, const int64_t pad_dim,
                            const int64_t embedding_size, const T* concat_out_grad_ptr,
                            T* dense_feature_grad) {
  for (int row = blockIdx.x; row < batch_size; row += gridDim.x) {
    for (int col = threadIdx.x; col < embedding_size; col += blockDim.x) {
      const int64_t out_offset = row * embedding_size + col;
      const int64_t in_offset = row * pad_dim * embedding_size + col;
      dense_feature_grad[out_offset] += concat_out_grad_ptr[in_offset];
    }
  }
}

template<typename T>
void SplitAddKernel(ep::Stream* stream, DataType data_type, int64_t batch_size, int64_t pad_dim,
                    int64_t concat_dim, int64_t embedding_size, const T* concat_out_grad_ptr,
                    T* dense_feature_grad, T* sparse_feature_grad) {
  // dense feature grad
  SliceAddGpu<T><<<GetNumBlocks(batch_size), kFusedInteractionBlockSize, 0,
                   stream->As<ep::CudaStream>()->hip_stream()>>>(
      batch_size, pad_dim, embedding_size, concat_out_grad_ptr, dense_feature_grad);

  // sparse feature grad
  auto primitive = ep::primitive::NewPrimitive<ep::primitive::CopyNdFactory>(DeviceType::kCUDA, 2);
  DimVector dst_shape = {batch_size, (concat_dim - 1) * embedding_size};
  DimVector dst_pos_vec = {0, 0};
  DimVector src_shape = {batch_size, pad_dim * embedding_size};
  DimVector src_pos_vec = {0, 1 * embedding_size};
  DimVector extent_vec = {batch_size, (concat_dim - 1) * embedding_size};
  primitive->Launch(stream, data_type, 2, sparse_feature_grad, dst_shape.data(), dst_pos_vec.data(),
                    concat_out_grad_ptr, src_shape.data(), src_pos_vec.data(), extent_vec.data());
}

int64_t GetPadDim(const int64_t dim) { return std::ceil(static_cast<float>(dim) / 8) * 8; }

template<typename T>
class FusedInteractionKernel final : public user_op::OpKernel {
 public:
  FusedInteractionKernel() = default;
  ~FusedInteractionKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dense_feature = ctx->Tensor4ArgNameAndIndex("dense_feature", 0);
    const user_op::Tensor* sparse_feature = ctx->Tensor4ArgNameAndIndex("sparse_feature", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    T* concat_out = ctx->Tensor4ArgNameAndIndex("concat_out", 0)->mut_dptr<T>();
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    CHECK_EQ(dense_feature->shape().NumAxes(), 2);
    CHECK_EQ(sparse_feature->shape().NumAxes(), 3);
    const int64_t batch_size = dense_feature->shape().At(0);
    const int64_t num_columns = sparse_feature->shape().At(1);
    const int64_t concat_dim = num_columns + 1;
    const int64_t embedding_size = dense_feature->shape().At(1);

    const int64_t pad_dim = GetPadDim(concat_dim);
    T* pad_tensor_ptr = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>());
    size_t pad_tensor_size =
        GetCudaAlignedSize(batch_size * (pad_dim - concat_dim) * embedding_size * sizeof(T));
    OF_CUDA_CHECK(hipMemsetAsync(pad_tensor_ptr, 0, pad_tensor_size,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));
    T* matmul_out = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + pad_tensor_size);
    size_t matmul_out_size = GetCudaAlignedSize(batch_size * pad_dim * pad_dim * sizeof(T));

    const int64_t tril_dim = num_columns * (num_columns + 1) / 2;
    const int64_t out_dim = out->shape().At(1);
    int32_t* tril_indices_ptr = reinterpret_cast<int32_t*>(tmp_buffer->mut_dptr<char>()
                                                           + pad_tensor_size + matmul_out_size);
    size_t tril_indices_size = GetCudaAlignedSize(tril_dim * sizeof(int32_t));
    GenerateTrilIndices(ctx->stream(), concat_dim, pad_dim, tril_indices_ptr);
    CHECK_LE(pad_tensor_size + matmul_out_size + tril_indices_size, tmp_buffer->shape().elem_cnt());

    std::vector<int64_t> in_cols;
    in_cols.push_back(1 * embedding_size);
    in_cols.push_back(sparse_feature->shape().At(1) * embedding_size);
    in_cols.push_back((pad_dim - concat_dim) * embedding_size);
    std::vector<const void*> in_ptrs;
    in_ptrs.push_back(dense_feature->dptr());
    in_ptrs.push_back(sparse_feature->dptr());
    in_ptrs.push_back(pad_tensor_ptr);
    // bsz, 32, 128
    ConcatKernel<T>(ctx->stream(), dense_feature->data_type(), batch_size, pad_dim * embedding_size,
                    in_cols, in_ptrs, reinterpret_cast<void*>(concat_out));
    // bsz, 32, 32
    BatchMatmul(ctx->stream(), dense_feature->data_type(), true, batch_size, pad_dim, pad_dim,
                embedding_size, concat_out, concat_out, matmul_out);

    GatherConcatKernel<T>(ctx->stream(), batch_size, out_dim, tril_dim, pad_dim, concat_dim,
                          embedding_size, tril_indices_ptr, matmul_out, dense_feature->dptr<T>(),
                          out->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
user_op::InferTmpSizeFn GenFusedInteractionInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    const user_op::TensorDesc& sparse_feature = ctx->InputTensorDesc("sparse_feature", 0);
    const int64_t batch_size = sparse_feature.shape().At(0);
    const int64_t num_columns = sparse_feature.shape().At(1);
    const int64_t concat_dim = num_columns + 1;
    const int64_t embedding_size = sparse_feature.shape().At(2);
    const int64_t pad_dim = GetPadDim(concat_dim);
    size_t pad_tensor_size =
        GetCudaAlignedSize(batch_size * (pad_dim - concat_dim) * embedding_size * sizeof(T));
    size_t matmul_out_size = GetCudaAlignedSize(batch_size * pad_dim * pad_dim * sizeof(T));
    size_t tril_indices_size =
        GetCudaAlignedSize(num_columns * (num_columns + 1) / 2 * sizeof(int32_t));
    return pad_tensor_size + matmul_out_size + tril_indices_size;
  };
}

#define REGISTER_FUSED_INTERACTION_KERNEL(dtype)                                                   \
  REGISTER_USER_KERNEL("fused_interaction")                                                        \
      .SetCreateFn<FusedInteractionKernel<dtype>>()                                                \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                             \
                       && (user_op::HobDataType("dense_feature", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn(GenFusedInteractionInferTmpSizeFn<dtype>());

REGISTER_FUSED_INTERACTION_KERNEL(float)
REGISTER_FUSED_INTERACTION_KERNEL(half)

template<typename T>
class FusedInteractionGradKernel final : public user_op::OpKernel {
 public:
  FusedInteractionGradKernel() = default;
  ~FusedInteractionGradKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* concat_out = ctx->Tensor4ArgNameAndIndex("concat_out", 0);
    user_op::Tensor* dense_feature_grad = ctx->Tensor4ArgNameAndIndex("dense_feature_grad", 0);
    user_op::Tensor* sparse_feature_grad = ctx->Tensor4ArgNameAndIndex("sparse_feature_grad", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t batch_size = dense_feature_grad->shape().At(0);
    const int64_t concat_dim = sparse_feature_grad->shape().At(1) + 1;
    const int64_t embedding_size = dense_feature_grad->shape().At(1);
    const int64_t pad_dim = GetPadDim(concat_dim);
    const int64_t out_dim = dy->shape().At(1);
    T* matmul_out_grad_ptr = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>());
    size_t matmul_out_grad_size = GetCudaAlignedSize(batch_size * pad_dim * pad_dim * sizeof(T));
    T* transposed_matmul_out_grad_ptr =
        reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + matmul_out_grad_size);
    size_t transposed_matmul_out_grad_size = matmul_out_grad_size;
    T* concat_out_grad_ptr = reinterpret_cast<T*>(
        tmp_buffer->mut_dptr<char>() + matmul_out_grad_size + transposed_matmul_out_grad_size);
    size_t concat_out_grad_size =
        GetCudaAlignedSize(batch_size * pad_dim * embedding_size * sizeof(T));
    CHECK_LE(matmul_out_grad_size + transposed_matmul_out_grad_size + concat_out_grad_size,
             tmp_buffer->shape().elem_cnt());
    ScatterSplitKernel(ctx->stream(), batch_size, out_dim, pad_dim, concat_dim, embedding_size,
                       dy->dptr<T>(), dense_feature_grad->mut_dptr<T>(), matmul_out_grad_ptr);
    const int64_t num_dims = 3;
    DimVector transpose_dims = {batch_size, pad_dim, pad_dim};
    std::vector<int32_t> perm = {0, 2, 1};
    const int64_t count = batch_size * pad_dim * pad_dim;
    auto transpose_primitive =
        ep::primitive::NewPrimitive<ep::primitive::PermuteFactory>(DeviceType::kCUDA, 3);
    transpose_primitive->Launch(ctx->stream(), dy->data_type(), num_dims, transpose_dims.data(),
                                matmul_out_grad_ptr, perm.data(), transposed_matmul_out_grad_ptr);

    auto add_primitive =
        ep::primitive::NewPrimitive<ep::primitive::AddFactory>(DeviceType::kCUDA, dy->data_type());
    add_primitive->Launch(ctx->stream(), matmul_out_grad_ptr, transposed_matmul_out_grad_ptr,
                          matmul_out_grad_ptr, count);
    BatchMatmul(ctx->stream(), dense_feature_grad->data_type(), false, batch_size, pad_dim,
                embedding_size, pad_dim, matmul_out_grad_ptr, concat_out->dptr<T>(),
                concat_out_grad_ptr);

    SplitAddKernel<T>(ctx->stream(), dense_feature_grad->data_type(), batch_size, pad_dim,
                      concat_dim, embedding_size, concat_out_grad_ptr,
                      dense_feature_grad->mut_dptr<T>(), sparse_feature_grad->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
user_op::InferTmpSizeFn GenFusedInteractionGradInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    const user_op::TensorDesc& sparse_feature_grad = ctx->InputTensorDesc("sparse_feature_grad", 0);
    const int64_t batch_size = sparse_feature_grad.shape().At(0);
    const int64_t concat_dim = sparse_feature_grad.shape().At(1) + 1;
    const int64_t embedding_size = sparse_feature_grad.shape().At(2);
    const int64_t pad_dim = GetPadDim(concat_dim);
    size_t matmul_out_grad_size = GetCudaAlignedSize(batch_size * pad_dim * pad_dim * sizeof(T));
    size_t transposed_matmul_out_grad_size = matmul_out_grad_size;
    size_t concat_out_grad_size =
        GetCudaAlignedSize(batch_size * pad_dim * embedding_size * sizeof(T));

    return matmul_out_grad_size + transposed_matmul_out_grad_size + concat_out_grad_size;
  };
}

#define REGISTER_FUSED_INTERACTION_GRAD_KERNEL(dtype)                                   \
  REGISTER_USER_KERNEL("fused_interaction_grad")                                        \
      .SetCreateFn<FusedInteractionGradKernel<dtype>>()                                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn(GenFusedInteractionGradInferTmpSizeFn<dtype>());

REGISTER_FUSED_INTERACTION_GRAD_KERNEL(float)
REGISTER_FUSED_INTERACTION_GRAD_KERNEL(half)

}  // namespace oneflow
