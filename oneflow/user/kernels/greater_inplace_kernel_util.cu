#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/user/kernels/greater_inplace_kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void GreaterInplacForwardGpu(const int64_t n, const T* x, const T* y, T* out) {
  CUDA_1D_KERNEL_LOOP_T(int64_t, i, n) {
    out[i] = x[i] > y[i] ? static_cast<T>(1) : static_cast<T>(0);
  }
}

template<typename T>
__global__ void ScalarGreaterInplacForwardGpu(const int64_t n, const T* x, const Scalar* operand,
                                              T* out) {
  CUDA_1D_KERNEL_LOOP_T(int64_t, i, n) {
    out[i] = x[i] > operand->As<T>() ? static_cast<T>(1) : static_cast<T>(0);
  }
}

template<>
__global__ void ScalarGreaterInplacForwardGpu<half>(const int64_t n, const half* x,
                                                    const Scalar* operand, half* out) {
  CUDA_1D_KERNEL_LOOP_T(int64_t, i, n) {
    out[i] =
        x[i] > __float2half(operand->As<float>()) ? static_cast<half>(1) : static_cast<half>(0);
  }
}

}  // namespace

template<typename T>
struct GreaterInplaceKernelUtil<DeviceType::kCUDA, T> {
  static void Forward(ep::Stream* stream, const int64_t n, const T* x, const T* y, T* out) {
    RUN_CUDA_KERNEL((GreaterInplacForwardGpu<T>), stream, n, n, x, y, out);
  }

  static void ScalarForward(ep::Stream* stream, const int64_t n, const T* x, const Scalar* operand,
                            T* out) {
    RUN_CUDA_KERNEL((ScalarGreaterInplacForwardGpu<T>), stream, n, n, x, operand, out);
  }
};

#define INSTANTIATE_GREATER_INPLACE_KERNEL_UTIL_CUDA(cpp_data_type, data_type) \
  template struct GreaterInplaceKernelUtil<DeviceType::kCUDA, cpp_data_type>;

OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GREATER_INPLACE_KERNEL_UTIL_CUDA,
                     FLOATING_DATA_TYPE_SEQ SIGNED_INT_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ)

#undef INSTANTIATE_GREATER_INPLACE_KERNEL_UTIL_CUDA

}  // namespace oneflow
