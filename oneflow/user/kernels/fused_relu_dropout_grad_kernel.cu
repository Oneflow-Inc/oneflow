#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include <hip/hip_runtime.h>
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/cuda/elementwise.cuh"

namespace oneflow {

namespace {

constexpr int32_t kWarpSize = 32;

template<typename T, typename IndexType, int pack_size, bool tail>
__global__ void VectorizedReluDropoutBitmaskBackwardKernel(
    const IndexType elem_cnt, const IndexType cols, const IndexType aux_ld, const float scale,
    const IndexType n_tail, const IndexType tail_offset, const T* dy, const int32_t* mask, T* dx) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  using LoadStoreType = cuda::elementwise::PackType<T, pack_size>;
  using LoadStorePack = cuda::elementwise::Pack<T, pack_size>;

  T t_scale = static_cast<T>(scale);
  for (IndexType linear_pack_index = global_thread_id * pack_size; linear_pack_index < elem_cnt;
       linear_pack_index += gridDim.x * blockDim.x * pack_size) {
    const LoadStoreType* dy_load = reinterpret_cast<const LoadStoreType*>(dy + linear_pack_index);
    LoadStorePack dy_vec;
    dy_vec.storage = *dy_load;

    LoadStorePack dx_vec;
#pragma unroll
    for (int i = 0; i < pack_size; i++) {
      const IndexType linear_index = (linear_pack_index + i);
      const IndexType row = linear_index / cols;
      const IndexType col = linear_index - row * cols;
      const int32_t col_mod_warpsize = col % kWarpSize;
      const IndexType aux_idx = ((row * aux_ld) + col) / kWarpSize;
      bool is_positive = mask[aux_idx] & (1 << col_mod_warpsize);
      dx_vec.elem[i] =
          dy_vec.elem[i] * static_cast<T>(static_cast<float>(is_positive)) * static_cast<T>(scale);
    }
    *(reinterpret_cast<LoadStoreType*>(dx + linear_pack_index)) = dx_vec.storage;
  }

  if (tail && global_thread_id < n_tail) {
    const IndexType tail_index = tail_offset + global_thread_id;
    const IndexType tail_row = tail_index / cols;
    const IndexType tail_col = tail_index - tail_row * cols;
    const IndexType tail_col_mod_warpsize = tail_col % kWarpSize;
    const IndexType tail_aux_idx = ((tail_row * aux_ld) + tail_col) / kWarpSize;
    bool is_positive = mask[tail_aux_idx] & (1 << tail_col_mod_warpsize);
    dx[tail_index] =
        dy[tail_index] * static_cast<T>(static_cast<float>(is_positive)) * static_cast<T>(scale);
  }
}

template<typename T>
void LaunchVectorizedReluDropoutBackwardKernel(ep::Stream* stream, const int64_t elem_cnt,
                                               const int64_t cols, const int64_t aux_ld,
                                               float scale, const T* dy, const int32_t* mask,
                                               T* dx) {
  constexpr int pack_size = cuda::elementwise::PackSize<T>();
  const int64_t pack_num = elem_cnt / pack_size;
  const int64_t tail_offset = pack_num * pack_size;
  const int64_t n_tail = elem_cnt - tail_offset;
  const bool tail = n_tail > 0 ? true : false;
  if (tail) {
    if (elem_cnt < GetMaxVal<int32_t>()) {
      stream->As<ep::CudaStream>()->LaunchKernelDefaultWaves(
          (VectorizedReluDropoutBitmaskBackwardKernel<T, int32_t, pack_size, true>),
          std::max<int64_t>(1, pack_num), elem_cnt, cols, aux_ld, scale, n_tail, tail_offset, dy,
          mask, dx);
    } else {
      stream->As<ep::CudaStream>()->LaunchKernelDefaultWaves(
          (VectorizedReluDropoutBitmaskBackwardKernel<T, int64_t, pack_size, true>),
          std::max<int64_t>(1, pack_num), elem_cnt, cols, aux_ld, scale, n_tail, tail_offset, dy,
          mask, dx);
    }
  } else {
    if (elem_cnt < GetMaxVal<int32_t>()) {
      stream->As<ep::CudaStream>()->LaunchKernelDefaultWaves(
          (VectorizedReluDropoutBitmaskBackwardKernel<T, int32_t, pack_size, false>),
          std::max<int64_t>(1, pack_num), elem_cnt, cols, aux_ld, scale, /*n_tail=*/0, tail_offset,
          dy, mask, dx);
    } else {
      stream->As<ep::CudaStream>()->LaunchKernelDefaultWaves(
          (VectorizedReluDropoutBitmaskBackwardKernel<T, int64_t, pack_size, false>),
          std::max<int64_t>(1, pack_num), elem_cnt, cols, aux_ld, scale, /*n_tail=*/0, tail_offset,
          dy, mask, dx);
    }
  }
}

template<typename T>
class FusedReluDropoutGradKernel final : public user_op::OpKernel,
                                         public user_op::CudaGraphSupport {
 public:
  FusedReluDropoutGradKernel() = default;
  ~FusedReluDropoutGradKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* mask = ctx->Tensor4ArgNameAndIndex("mask", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const float scale = ctx->Attr<float>("scale");

    const int64_t cols = dy->shape_view().At(1);
    const int64_t aux_ld = mask->shape_view().At(1) * 32;
    const int64_t elem_cnt = dy->shape_view().elem_cnt();
    LaunchVectorizedReluDropoutBackwardKernel<T>(
        ctx->stream(), elem_cnt, cols, aux_ld, scale, reinterpret_cast<const T*>(dy->dptr()),
        mask->dptr<int32_t>(), reinterpret_cast<T*>(dx->mut_dptr()));
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_RELU_DROPOUT_GRAD_KERNEL_GPU(cpp_type, data_type) \
  REGISTER_USER_KERNEL("fused_relu_dropout_grad")                        \
      .SetCreateFn<FusedReluDropoutGradKernel<cpp_type>>()               \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)   \
                       && (user_op::HobDataType("dx", 0) == data_type));

REGISTER_FUSED_RELU_DROPOUT_GRAD_KERNEL_GPU(float, DataType::kFloat)
REGISTER_FUSED_RELU_DROPOUT_GRAD_KERNEL_GPU(half, DataType::kFloat16)
#if CUDA_VERSION >= 11000
REGISTER_FUSED_RELU_DROPOUT_GRAD_KERNEL_GPU(hip_bfloat16, DataType::kBFloat16)
#endif

}  // namespace

}  // namespace oneflow
