#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/sparse_softmax_cross_entropy_kernel_util.cuh"
// #include "oneflow/user/kernels/sparse_softmax_cross_entropy_kernel_util.h"
#include "oneflow/core/ndarray/xpu_var_ndarray.h"

namespace oneflow {
namespace user_op {
namespace {

template<typename T, typename K>
__global__ void ComputeDiffGpu(const int64_t num_instances, const int64_t num_classes,
                               const int64_t depth, const int64_t lower_bound, const T* prob,
                               const K* labels, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, num_instances) {
    const int32_t row_id = i / num_classes;
    const int32_t col_id = i - row_id * num_classes;
    assert(labels[row_id] >= 0);
    assert(labels[row_id] < depth);
    K label = labels[row_id] - lower_bound;
    if (label == col_id) {
      dx[i] = dy[row_id] * (prob[i] - 1);
    } else {
      dx[i] = dy[row_id] * prob[i];
    }
  }
}

template<typename K>
__global__ void ComputeDiffGpuHalf(const int64_t num_instances, const int64_t num_classes,
                                   const int64_t depth, const int64_t lower_bound, const half* prob,
                                   const K* labels, const half* dy, half* dx) {
  CUDA_1D_KERNEL_LOOP(i, num_instances) {
    const int32_t row_id = i / num_classes;
    const int32_t col_id = i - row_id * num_classes;
    assert(labels[row_id] >= 0);
    assert(labels[row_id] < depth);
    K label = labels[row_id] - lower_bound;
    if (label == col_id) {
      dx[i] = __hmul(dy[row_id], __hsub(prob[i], __float2half(1.0)));
    } else {
      dx[i] = __hmul(dy[row_id], prob[i]);
    }
  }
}

template<typename T, typename K>
void ComputeFront(DeviceCtx* ctx, const int64_t num_instances, const int64_t num_classes,
                  const T* in, T* prob, T* y, void* temp_storage, const size_t temp_storage_bytes,
                  const MemoryCase& prob_mem_case, const MemoryCase& tmp_buffer_mem_case) {
  auto Val = NdarrayUtil<DeviceType::kGPU, T>::GetValNdarrayBuilder();
  auto Var = NdarrayUtil<DeviceType::kGPU, T>::GetVarNdarrayBuilder();

  const size_t min_temp_storage_bytes =
      SparseSoftmaxCrossEntropyTempStorageSize<T>(num_instances, num_classes);
  assert(temp_storage_bytes >= min_temp_storage_bytes);

  const size_t reduce_operation_size =
      SparseSoftmaxCrossEntropyReduceOperationSize<T>(num_instances, num_classes);
  const size_t sum_result_size = SparseSoftmaxCrossEntropySumResultSize<T>(num_instances);

  T* reduce_storage = reinterpret_cast<T*>(temp_storage);

  auto reduce_storage_var =
      Var({static_cast<int64_t>(reduce_operation_size / sizeof(T))}, reduce_storage);
  T* sum_result =
      reinterpret_cast<T*>(reinterpret_cast<unsigned char*>(temp_storage) + reduce_operation_size);
  T* sub_result = reinterpret_cast<T*>(reinterpret_cast<unsigned char*>(temp_storage)
                                       + reduce_operation_size + sum_result_size);

  // max | tmp[i] = Max_j(in[i][j])
  NdarrayUtil<DeviceType::kGPU, T>::ReduceMax(ctx, Var({num_instances, 1}, sum_result),
                                              Val({num_instances, num_classes}, in),
                                              reduce_storage_var);
  // sub | prob[i][j] = in[i][j] - tmp[i]
  NdarrayUtil<DeviceType::kGPU, T>::BroadcastSub(ctx, Var({num_instances, num_classes}, sub_result),
                                                 Val({num_instances, num_classes}, in),
                                                 Val({num_instances, 1}, sum_result));
  // exp | prob[i][j] = exp(prob[i][j])
  hipMemcpy(prob, sub_result, reduce_operation_size, hipMemcpyHostToDevice);
  // AutoMemcpy(ctx, prob, new_tmp, reduce_temp_storage_bytes, prob_mem_case,
  // tmp_buffer_mem_case);
  NdarrayUtil<DeviceType::kGPU, T>::InplaceExp(ctx, Var({num_instances, num_classes}, prob));
  // sum | tmp[i] = Sum_j(prob[i][j])
  NdarrayUtil<DeviceType::kGPU, T>::ReduceSum(ctx, Var({num_instances, 1}, sum_result),
                                              Val({num_instances, num_classes}, prob),
                                              reduce_storage_var);

  NdarrayUtil<DeviceType::kGPU, T>::InplaceBroadcastDiv(
      ctx, Var({num_instances, num_classes}, prob),
      Val({num_instances, 1}, sum_result));  // for backward
}
}  // namespace

template<typename T, typename K>
struct SparseSoftmaxCrossEntropyKernelUtil<DeviceType::kGPU, T, K> {
  static void Compute(DeviceCtx* ctx, const int64_t num_instances, const int64_t num_classes,
                      const int64_t depth, const int64_t lower_bound, const T* in, T* prob,
                      const K* labels, T* y, void* temp_storage, const size_t temp_storage_bytes,
                      const MemoryCase& prob_mem_case, const MemoryCase& tmp_buffer_mem_case) {
    ComputeFront<T, K>(ctx, num_instances, num_classes, in, prob, y, temp_storage,
                       temp_storage_bytes, prob_mem_case, tmp_buffer_mem_case);

    const size_t reduce_operation_size =
        SparseSoftmaxCrossEntropyReduceOperationSize<T>(num_instances, num_classes);
    const size_t sum_result_size = SparseSoftmaxCrossEntropySumResultSize<T>(num_instances);

    T* sum_result = reinterpret_cast<T*>(reinterpret_cast<unsigned char*>(temp_storage)
                                         + reduce_operation_size);
    T* sub_result = reinterpret_cast<T*>(reinterpret_cast<unsigned char*>(temp_storage)
                                         + reduce_operation_size + sum_result_size);

    ComputeSparseSoftmaxCrossEntropyResultGpu<T, K>
        <<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            num_instances, num_classes, depth, lower_bound, labels, sum_result, sub_result, y);
  }

  static void ComputeDiff(DeviceCtx* ctx, const int64_t num_instances, const int64_t num_classes,
                          const int64_t depth, const int64_t lower_bound, const T* prob,
                          const K* labels, const T* dy, T* dx) {
    ComputeDiffGpu<<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0,
                     ctx->hip_stream()>>>(num_instances, num_classes, depth, lower_bound, prob,
                                           labels, dy, dx);
  }
};

template<typename K>
struct SparseSoftmaxCrossEntropyKernelUtil<DeviceType::kGPU, float16, K> {
  static void Compute(DeviceCtx* ctx, const int64_t num_instances, const int64_t num_classes,
                      const int64_t depth, const int64_t lower_bound, const float16* in,
                      float16* prob, const K* labels, float16* y, void* temp_storage,
                      const size_t temp_storage_bytes, const MemoryCase& prob_mem_case,
                      const MemoryCase& tmp_buffer_mem_case) {
    ComputeFront<float16, K>(ctx, num_instances, num_classes, in, prob, y, temp_storage,
                             temp_storage_bytes, prob_mem_case, tmp_buffer_mem_case);

    const size_t reduce_operation_size =
        SparseSoftmaxCrossEntropyReduceOperationSize<float16>(num_instances, num_classes);
    const size_t sum_result_size = SparseSoftmaxCrossEntropySumResultSize<float16>(num_instances);
    float16* sum_result = reinterpret_cast<float16*>(reinterpret_cast<unsigned char*>(temp_storage)
                                                     + reduce_operation_size);
    float16* sub_result = reinterpret_cast<float16*>(reinterpret_cast<unsigned char*>(temp_storage)
                                                     + reduce_operation_size + sum_result_size);

    ComputeSparseSoftmaxCrossEntropyResultGpuHalf<K>
        <<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            num_instances, num_classes, depth, lower_bound, labels,
            reinterpret_cast<half*>(sum_result), reinterpret_cast<half*>(sub_result),
            reinterpret_cast<half*>(y));
  }

  static void ComputeDiff(DeviceCtx* ctx, const int64_t num_instances, const int64_t num_classes,
                          const int64_t depth, const int64_t lower_bound, const float16* prob,
                          const K* labels, const float16* dy, float16* dx) {
    ComputeDiffGpuHalf<<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0,
                         ctx->hip_stream()>>>(
        num_instances, num_classes, depth, lower_bound, reinterpret_cast<const half*>(prob), labels,
        reinterpret_cast<const half*>(dy), reinterpret_cast<half*>(dx));
  }
};

#define INSTANTIATE_SPARSE_SOFTMAX_CROSS_ENTROPY_KERNEL_UTIL_GPU(data_type_pair, index_type_pair) \
  template struct SparseSoftmaxCrossEntropyKernelUtil<                                            \
      DeviceType::kGPU, OF_PP_PAIR_FIRST(data_type_pair), OF_PP_PAIR_FIRST(index_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_SPARSE_SOFTMAX_CROSS_ENTROPY_KERNEL_UTIL_GPU,
                                 FLOATING_DATA_TYPE_SEQ FLOAT16_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ);
#undef INSTANTIATE_SPARSE_SOFTMAX_CROSS_ENTROPY_KERNEL_UTIL_GPU

}  // namespace user_op
}  // namespace oneflow
