#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/sparse_softmax_cross_entropy_kernel_util.h"
#include "oneflow/core/cuda/softmax.cuh"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {
namespace user_op {
namespace {

template<typename T>
__inline__ __device__ T Exp(T x);

template<>
__inline__ __device__ float Exp<float>(float x) {
#ifdef OF_SOFTMAX_USE_FAST_MATH
  return __expf(x);
#else
  return exp(x);
#endif
}

template<>
__inline__ __device__ double Exp<double>(double x) {
  return exp(x);
}

template<>
__inline__ __device__ half Exp<half>(half x) {
#ifdef OF_SOFTMAX_USE_FAST_MATH
  return __float2half(__expf(__half2float(x)));
#else
  return __float2half(exp(__half2float(x)));
#endif
}

template<typename T, typename K, typename IndexType>
__global__ void ComputeDiffGpu(const int64_t num_instances, const int64_t num_classes,
                               const int64_t depth, const int64_t lower_bound, const T* prob,
                               const K* labels, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP_T(IndexType, i, num_instances) {
    const IndexType row_id = i / num_classes;
    const IndexType col_id = i - row_id * num_classes;
    assert(labels[row_id] >= 0);
    assert(labels[row_id] < depth);
    K label = labels[row_id] - lower_bound;
    if (label == col_id) {
      dx[i] = dy[row_id] * (Exp(prob[i]) - 1);
    } else {
      dx[i] = dy[row_id] * Exp(prob[i]);
    }
  }
}

template<typename K, typename IndexType>
__global__ void ComputeDiffGpuHalf(const int64_t num_instances, const int64_t num_classes,
                                   const int64_t depth, const int64_t lower_bound, const half* prob,
                                   const K* labels, const half* dy, half* dx) {
  CUDA_1D_KERNEL_LOOP_T(IndexType, i, num_instances) {
    const IndexType row_id = i / num_classes;
    const IndexType col_id = i - row_id * num_classes;
    assert(labels[row_id] >= 0);
    assert(labels[row_id] < depth);
    K label = labels[row_id] - lower_bound;
    if (label == col_id) {
      dx[i] = __hmul(dy[row_id], __hsub(Exp(prob[i]), __float2half(1.0)));
    } else {
      dx[i] = __hmul(dy[row_id], Exp(prob[i]));
    }
  }
}

}  // namespace

template<typename T, typename K>
struct SparseSoftmaxCrossEntropyKernelUtil<DeviceType::kCUDA, T, K> {
  static void ComputeDiff(ep::Stream* stream, const int64_t num_instances,
                          const int64_t num_classes, const int64_t depth, const int64_t lower_bound,
                          const T* prob, const K* labels, const T* dy, T* dx) {
    if (num_instances < GetMaxVal<int32_t>() / 2) {
      ComputeDiffGpu<T, K, int32_t><<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock,
                                      0, stream->As<ep::CudaStream>()->hip_stream()>>>(
          num_instances, num_classes, depth, lower_bound, prob, labels, dy, dx);
    } else {
      // NOTE(chengcheng): int division ('/') of i will reduce performance of int64_t.
      ComputeDiffGpu<T, K, int64_t><<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock,
                                      0, stream->As<ep::CudaStream>()->hip_stream()>>>(
          num_instances, num_classes, depth, lower_bound, prob, labels, dy, dx);
    }
  }
};

template<typename K>
struct SparseSoftmaxCrossEntropyKernelUtil<DeviceType::kCUDA, float16, K> {
  static void ComputeDiff(ep::Stream* stream, const int64_t num_instances,
                          const int64_t num_classes, const int64_t depth, const int64_t lower_bound,
                          const float16* prob, const K* labels, const float16* dy, float16* dx) {
    if (num_instances < GetMaxVal<int32_t>() / 2) {
      ComputeDiffGpuHalf<K, int32_t><<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock,
                                       0, stream->As<ep::CudaStream>()->hip_stream()>>>(
          num_instances, num_classes, depth, lower_bound, reinterpret_cast<const half*>(prob),
          labels, reinterpret_cast<const half*>(dy), reinterpret_cast<half*>(dx));
    } else {
      ComputeDiffGpuHalf<K, int64_t><<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock,
                                       0, stream->As<ep::CudaStream>()->hip_stream()>>>(
          num_instances, num_classes, depth, lower_bound, reinterpret_cast<const half*>(prob),
          labels, reinterpret_cast<const half*>(dy), reinterpret_cast<half*>(dx));
    }
  }
};

#define INSTANTIATE_SPARSE_SOFTMAX_CROSS_ENTROPY_KERNEL_UTIL_CUDA(data_type_pair, index_type_pair) \
  template struct SparseSoftmaxCrossEntropyKernelUtil<                                             \
      DeviceType::kCUDA, OF_PP_PAIR_FIRST(data_type_pair), OF_PP_PAIR_FIRST(index_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_SPARSE_SOFTMAX_CROSS_ENTROPY_KERNEL_UTIL_CUDA,
                                 FLOATING_DATA_TYPE_SEQ FLOAT16_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ);
#undef INSTANTIATE_SPARSE_SOFTMAX_CROSS_ENTROPY_KERNEL_UTIL_CUDA

}  // namespace user_op
}  // namespace oneflow
