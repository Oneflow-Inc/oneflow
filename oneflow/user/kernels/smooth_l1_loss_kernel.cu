#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/loss_kernel_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {
namespace user_op {

namespace {

using namespace loss;

template<typename T>
__global__ void ComputeSmoothL1Out(int64_t elem_cnt, const T* input, const T* target, T* out,
                                   const float beta, const float inv_beta) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const T abs_diff = abs(input[i] - target[i]);
    if (abs_diff < beta) {
      out[i] = 0.5 * abs_diff * abs_diff * inv_beta;
    } else {
      out[i] = abs_diff - 0.5 * beta;
    }
  }
}

template<>
__global__ void ComputeSmoothL1Out(int64_t elem_cnt, const half* input, const half* target,
                                   half* out, const float beta, const float inv_beta) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  const half half_zero = __float2half(0.0);
  const half half_one = __float2half(0.5);
  const half half_beta = __float2half(beta);
  const half half_inv_beta = __float2half(inv_beta);
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const half diff = __hsub(input[i], target[i]);
    const half abs_diff = __hlt(diff, half_zero) ? __hneg(diff) : diff;
    if (__hlt(abs_diff, half_beta)) {
      out[i] = __hmul(__hmul(half_one, abs_diff), __hmul(abs_diff, half_inv_beta));
    } else {
      out[i] = __hsub(abs_diff, __hmul(half_one, half_beta));
    }
  }
#else
  printf("use half need nvcc arch >= 530");
  assert(false);
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)*/
}

template<typename T>
__global__ void ComputeSmoothL1GradOut(int64_t elem_cnt, float inv_elem_cnt, const T* input,
                                       const T* target, const T* dy, T* dx,
                                       const ReductionType reduction_type, const float beta,
                                       const float inv_beta) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const T diff = input[i] - target[i];
    const T abs_diff = abs(diff);
    T dx_val;
    if (abs_diff < beta) {
      dx_val = diff * inv_beta;
    } else {
      dx_val = (diff > GetZeroVal<T>()) - (diff < GetZeroVal<T>());
    }
    const T dy_val = reduction_type == ReductionType::kNone ? dy[i] : *dy;
    dx_val = dx_val * dy_val;
    if (reduction_type == ReductionType::kMean) { dx_val *= inv_elem_cnt; };
    dx[i] = dx_val;
  }
}

template<>
__global__ void ComputeSmoothL1GradOut(int64_t elem_cnt, float inv_elem_cnt, const half* input,
                                       const half* target, const half* dy, half* dx,
                                       const ReductionType reduction_type, const float beta,
                                       const float inv_beta) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  const half half_zero = __float2half(0.0);
  const half half_one = __float2half(1.0);
  const half half_beta = __float2half(beta);
  const half half_inv_beta = __float2half(inv_beta);
  const half half_inv_elem_cnt = __float2half(inv_elem_cnt);

  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const half diff = __hsub(input[i], target[i]);
    const half abs_diff = __hlt(diff, half_zero) ? __hneg(diff) : diff;
    half dx_val;
    if (__hlt(abs_diff, half_beta)) {
      dx_val = __hmul(diff, half_inv_beta);
    } else {
      const half left = __hgt(diff, half_zero) ? half_one : half_zero;
      const half right = __hlt(diff, half_zero) ? half_one : half_zero;
      dx_val = __hsub(left, right);
    }
    const half dy_val = reduction_type == ReductionType::kNone ? dy[i] : *dy;
    dx_val = __hmul(dx_val, dy_val);
    if (reduction_type == ReductionType::kMean) { dx_val = __hmul(dx_val, half_inv_elem_cnt); };
    dx[i] = dx_val;
  }
#else
  printf("use half need nvcc arch >= 530");
  assert(false);
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)*/
}

template<typename T>
class SmoothL1LossKernel : public SimpleLossKernel<DeviceType::kCUDA, T, SmoothL1LossKernel<T>> {
 public:
  void ComputeOut(user_op::KernelComputeContext* ctx, int64_t elem_cnt, const T* input,
                  const T* target, T* out) const {
    const float beta = ctx->Attr<float>("beta");
    ComputeSmoothL1Out<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                         ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        elem_cnt, input, target, out, beta, static_cast<float>(1.0 / beta));
  }
};

template<typename T>
class SmoothL1LossGradKernel
    : public SimpleLossGradKernel<DeviceType::kCUDA, T, SmoothL1LossGradKernel<T>> {
 public:
  void ComputeOut(user_op::KernelComputeContext* ctx, int64_t elem_cnt, const T* input,
                  const T* target, const T* dy, T* dx, const ReductionType reduction) const {
    const float beta = ctx->Attr<float>("beta");
    ComputeSmoothL1GradOut<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                             ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        elem_cnt, static_cast<float>(1.0 / elem_cnt), input, target, dy, dx, reduction, beta,
        static_cast<float>(1.0 / beta));
  }
};

}  // namespace

REGISTER_SIMPLE_LOSS_KERNEL_CUDA("smooth_l1_loss", SmoothL1LossKernel)
REGISTER_SIMPLE_LOSS_GRAD_KERNEL_CUDA("smooth_l1_loss_grad", SmoothL1LossGradKernel)

}  // namespace user_op
}  // namespace oneflow
