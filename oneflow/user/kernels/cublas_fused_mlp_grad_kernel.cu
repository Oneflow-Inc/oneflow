/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/user/kernels/cublas_fused_mlp_util.cuh"
// CUBLAS_AUX_EPILOGUE only support in cuda11.4 or higher version, in cuda11.4 it need static link.
#if CUDA_VERSION >= 11060

namespace oneflow {

namespace {

class MatmulGradKernelState final : public user_op::OpKernelState {
 public:
  MatmulGradKernelState() {
    OF_CUDA_CHECK(hipStreamCreate(&cuda_stream_));
    OF_CUBLAS_CHECK(hipblasLtCreate(&cublas_lt_handle_));
    OF_CUDA_CHECK(hipMalloc(&workspace_, 8 * 1024 * 1024));
  }
  ~MatmulGradKernelState() {
    OF_CUDA_CHECK(hipStreamSynchronize(cuda_stream_));
    OF_CUBLAS_CHECK(hipblasLtDestroy(cublas_lt_handle_));
    OF_CUDA_CHECK(hipStreamDestroy(cuda_stream_));
    OF_CUDA_CHECK(hipFree(workspace_));
  }
  hipStream_t hip_stream() const { return cuda_stream_; }
  hipblasLtHandle_t cublas_lt_handle() const { return cublas_lt_handle_; }
  size_t cublas_workspace_size() const { return 8 * 1024 * 1024; }
  void* cublas_workspace() const { return workspace_; }

 private:
  hipStream_t cuda_stream_{};
  hipblasLtHandle_t cublas_lt_handle_{};
  void* workspace_{};
};

template<typename T>
class CublasFusedMLPGradKernel final : public user_op::OpKernel,
                                                public user_op::CudaGraphSupport {
 public:
  CublasFusedMLPGradKernel() {
    OF_CUDA_CHECK(hipEventCreate(&main_stream_event));
    OF_CUDA_CHECK(hipEventCreate(&async_weight_grad_event));
  };
  ~CublasFusedMLPGradKernel() override {
    OF_CUDA_CHECK(hipEventDestroy(main_stream_event));
    OF_CUDA_CHECK(hipEventDestroy(async_weight_grad_event));
  };

  std::shared_ptr<user_op::OpKernelCache> InitOpKernelCache(
      user_op::KernelCacheContext* ctx) const override {
    return CreateCublasFusedMLPKernelCache();
  }

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<MatmulGradKernelState>();
  }

 private:
  hipEvent_t main_stream_event;
  hipEvent_t async_weight_grad_event;

  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache* cache) const override {
    printf("Here enter kernel \n"); 
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    void* dy_tmp_buf = tmp_buffer->mut_dptr();
    size_t offset = 0;             
    // const user_op::Tensor* weight = ctx->Tensor4ArgNameAndIndex("weight", 0);
    // const user_op::Tensor* aux = ctx->Tensor4ArgNameAndIndex("aux", 0);
    // const user_op::Tensor* hidden = ctx->Tensor4ArgNameAndIndex("hidden", 0);

    // user_op::Tensor* d_bias = ctx->Tensor4ArgNameAndIndex("d_bias", 0);
    user_op::Tensor* d_grad = ctx->Tensor4ArgNameAndIndex("d_grad", 0);
    // user_op::Tensor* d_weight = ctx->Tensor4ArgNameAndIndex("d_weight", 0);
    
    const int64_t weight_size = ctx->input_size("weight"); 

    const auto* matmul_grad_cache =
        CHECK_NOTNULL(dynamic_cast<const CublasFusedMLPKernelCache*>(cache));
    auto* hip_stream = ctx->stream()->As<ep::CudaStream>();

    auto* kernel_state = dynamic_cast<MatmulGradKernelState*>(state);

    const DataType data_type = dy->data_type();
    const hipblasComputeType_t cublas_compute_dtype = GetComputeType(data_type);
    const hipDataType cuda_data_type = GetCudaDataType(data_type);
    size_t cublas_m = 0, cublas_n = 0, cublas_k = 0;
    int64_t cublas_lda = 0, cublas_ldb = 0, cublas_ldc = 0;

    double alpha = 1.0;
    auto sp_alpha = GetCublasScalarParameter(alpha, cublas_compute_dtype);
    double beta = 0.0;
    auto sp_beta = GetCublasScalarParameter(beta, cublas_compute_dtype);

    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT; // = CUBLASLT_EPILOGUE_DRELU_BGRAD

    // currently only support 2D matmul.
    DimVector dy_shape(2);
    DimVector weight_shape(2);
    DimVector hidden_shape(2);
    dy->shape().ToDimVector(&dy_shape);
    const void* dgrad_buf = dy->dptr(); 

    for(int idx = weight_size - 1; idx > -1; idx--){
        if(idx != 0){
            const user_op::Tensor* weight = ctx->Tensor4ArgNameAndIndex("weight", idx);
            printf("weight shape is: %ld, %ld \n", weight->shape().At(0), weight->shape().At(1)); 
            const user_op::Tensor* aux = ctx->Tensor4ArgNameAndIndex("aux", idx-1);
            printf("aux shape is: %ld, %ld \n", aux->shape().At(0), aux->shape().At(1)); 
            user_op::Tensor* d_bias = ctx->Tensor4ArgNameAndIndex("d_bias", idx-1);
            printf("dbias shape is: %ld\n", d_bias->shape().At(0)); 

            weight->shape().ToDimVector(&weight_shape);
            epilogue = CUBLASLT_EPILOGUE_DRELU_BGRAD; 
            InferMatmulCublasMNK(dy_shape, weight_shape,
                                /*transpose_a=*/ep::primitive::BlasTransposeType::N,
                                /*transpose_b=*/ep::primitive::BlasTransposeType::N, &cublas_m, &cublas_n,
                                &cublas_k, &cublas_lda, &cublas_ldb, &cublas_ldc);
            SetCublasAttr(matmul_grad_cache, cublas_compute_dtype, cuda_data_type, /*need_aux=*/true,
                        /*transpose_a=*/ep::primitive::BlasTransposeType::N,
                        /*transpose_b=*/ep::primitive::BlasTransposeType::N, epilogue, d_bias->mut_dptr(),
                        aux->dptr(), cublas_m, cublas_n, cublas_k, cublas_lda, cublas_ldb, cublas_ldc);
            /*
            a = dy, b = weight
            cublas_a=weight, cublas_b=dy
            */
            OF_CUDA_CHECK(hipEventRecord(main_stream_event, hip_stream->hip_stream()));
            OF_CUBLAS_CHECK(
                hipblasLtMatmul(hip_stream->cublas_lt_handle(), matmul_grad_cache->operation_desc,
                            &sp_alpha, weight->dptr(), matmul_grad_cache->cublas_a_desc, dgrad_buf,
                            matmul_grad_cache->cublas_b_desc, &sp_beta, dy_tmp_buf,
                            matmul_grad_cache->cublas_c_desc, dy_tmp_buf,
                            matmul_grad_cache->cublas_c_desc, nullptr, hip_stream->cublas_workspace(),
                            hip_stream->cublas_workspace_size(), hip_stream->hip_stream()));
            OF_CUDA_CHECK(hipDeviceSynchronize()); 
        } else {
          const user_op::Tensor* weight = ctx->Tensor4ArgNameAndIndex("weight", 0);
          printf("weight shape is: %ld, %ld \n", weight->shape().At(0), weight->shape().At(1)); 

          weight->shape().ToDimVector(&weight_shape);
          epilogue = HIPBLASLT_EPILOGUE_DEFAULT; 
          InferMatmulCublasMNK(dy_shape, weight_shape,
                              /*transpose_a=*/ep::primitive::BlasTransposeType::N,
                              /*transpose_b=*/ep::primitive::BlasTransposeType::N, &cublas_m, &cublas_n,
                              &cublas_k, &cublas_lda, &cublas_ldb, &cublas_ldc);
          SetCublasAttr(matmul_grad_cache, cublas_compute_dtype, cuda_data_type, /*need_aux=*/false,
                        /*transpose_a=*/ep::primitive::BlasTransposeType::N,
                        /*transpose_b=*/ep::primitive::BlasTransposeType::N, epilogue, nullptr,
                        nullptr, cublas_m, cublas_n, cublas_k, cublas_lda, cublas_ldb, cublas_ldc);
          /*
          a = dy, b = weight
          cublas_a=weight, cublas_b=dy
          */
          OF_CUDA_CHECK(hipEventRecord(main_stream_event, hip_stream->hip_stream()));
          OF_CUBLAS_CHECK(
              hipblasLtMatmul(hip_stream->cublas_lt_handle(), matmul_grad_cache->operation_desc,
                          &sp_alpha, weight->dptr(), matmul_grad_cache->cublas_a_desc, dgrad_buf,
                          matmul_grad_cache->cublas_b_desc, &sp_beta, d_grad->mut_dptr(),
                          matmul_grad_cache->cublas_c_desc, d_grad->mut_dptr(),
                          matmul_grad_cache->cublas_c_desc, nullptr, hip_stream->cublas_workspace(),
                          hip_stream->cublas_workspace_size(), hip_stream->hip_stream()));
          OF_CUDA_CHECK(hipDeviceSynchronize()); 
        }
        alpha = 1.0;
        sp_alpha = GetCublasScalarParameter(alpha, cublas_compute_dtype);
        beta = 0.0;
        sp_beta = GetCublasScalarParameter(beta, cublas_compute_dtype);

        // currently only support 2D matmul.
        if(idx != 0){
          const user_op::Tensor* hidden = ctx->Tensor4ArgNameAndIndex("hidden", idx-1); // here
          user_op::Tensor* d_weight = ctx->Tensor4ArgNameAndIndex("d_weight", idx);
          printf("hidden shape is: %ld, %ld \n", hidden->shape().At(0), hidden->shape().At(1)); 
          printf("d_weight shape is: %ld, %ld \n", d_weight->shape().At(0), d_weight->shape().At(1)); 

          hidden->shape().ToDimVector(&hidden_shape);

          epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

          InferMatmulCublasMNK(dy_shape, hidden_shape,
                              /*transpose_a=*/ep::primitive::BlasTransposeType::T,
                              /*transpose_b=*/ep::primitive::BlasTransposeType::N, &cublas_m, &cublas_n,
                              &cublas_k, &cublas_lda, &cublas_ldb, &cublas_ldc);

          SetCublasAttr(matmul_grad_cache, cublas_compute_dtype, cuda_data_type, /*need_aux=*/false,
                        /*transpose_a=*/ep::primitive::BlasTransposeType::T,
                        /*transpose_b=*/ep::primitive::BlasTransposeType::N, epilogue, nullptr, nullptr,
                        cublas_m, cublas_n, cublas_k, cublas_lda, cublas_ldb, cublas_ldc);
          OF_CUDA_CHECK(hipStreamWaitEvent(kernel_state->hip_stream(), main_stream_event));
          OF_CUBLAS_CHECK(
              hipblasLtMatmul(kernel_state->cublas_lt_handle(), matmul_grad_cache->operation_desc,
                          &sp_alpha, hidden->dptr(), matmul_grad_cache->cublas_a_desc, dgrad_buf,
                          matmul_grad_cache->cublas_b_desc, &sp_beta, d_weight->mut_dptr(),
                          matmul_grad_cache->cublas_c_desc, d_weight->mut_dptr(),
                          matmul_grad_cache->cublas_c_desc, nullptr, kernel_state->cublas_workspace(),
                          kernel_state->cublas_workspace_size(), kernel_state->hip_stream()));
          OF_CUDA_CHECK(hipEventRecord(async_weight_grad_event, kernel_state->hip_stream()));
          OF_CUDA_CHECK(hipStreamWaitEvent(hip_stream->hip_stream(), async_weight_grad_event));
          OF_CUDA_CHECK(hipDeviceSynchronize()); 
          
          // compute dy shape
          dy_shape.at(1) = weight_shape.at(1);
          // compute dybuf
          dgrad_buf = dy_tmp_buf; 
          // if(idx != 0){
            offset += GetCudaAlignedSize(dy_shape.at(0) * dy_shape.at(1) * sizeof(T)); 
            printf("Offset size is: %ld \n", dy_shape.at(0) * dy_shape.at(1)); 
            dy_tmp_buf = reinterpret_cast<void*>(tmp_buffer->mut_dptr<char>() + offset);
          // }
        } else {
          user_op::Tensor* d_weight = ctx->Tensor4ArgNameAndIndex("d_weight", 0);
          x->shape().ToDimVector(&hidden_shape);
          epilogue = HIPBLASLT_EPILOGUE_DEFAULT;
          InferMatmulCublasMNK(dy_shape, hidden_shape,
                              /*transpose_a=*/ep::primitive::BlasTransposeType::T,
                              /*transpose_b=*/ep::primitive::BlasTransposeType::N, &cublas_m, &cublas_n,
                              &cublas_k, &cublas_lda, &cublas_ldb, &cublas_ldc);
          SetCublasAttr(matmul_grad_cache, cublas_compute_dtype, cuda_data_type, /*need_aux=*/false,
                        /*transpose_a=*/ep::primitive::BlasTransposeType::T,
                        /*transpose_b=*/ep::primitive::BlasTransposeType::N, epilogue, nullptr, nullptr,
                        cublas_m, cublas_n, cublas_k, cublas_lda, cublas_ldb, cublas_ldc);
          OF_CUDA_CHECK(hipStreamWaitEvent(kernel_state->hip_stream(), main_stream_event));
          OF_CUBLAS_CHECK(
              hipblasLtMatmul(kernel_state->cublas_lt_handle(), matmul_grad_cache->operation_desc,
                          &sp_alpha, x->dptr(), matmul_grad_cache->cublas_a_desc, dgrad_buf,
                          matmul_grad_cache->cublas_b_desc, &sp_beta, d_weight->mut_dptr(),
                          matmul_grad_cache->cublas_c_desc, d_weight->mut_dptr(),
                          matmul_grad_cache->cublas_c_desc, nullptr, kernel_state->cublas_workspace(),
                          kernel_state->cublas_workspace_size(), kernel_state->hip_stream()));
          OF_CUDA_CHECK(hipEventRecord(async_weight_grad_event, kernel_state->hip_stream()));
          OF_CUDA_CHECK(hipStreamWaitEvent(hip_stream->hip_stream(), async_weight_grad_event));
          OF_CUDA_CHECK(hipDeviceSynchronize()); 
        }
    }
    printf("end kernel \n"); 
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUBLAS_FUSED_MLP_GRAD_KERNEL(dtype)        \
  REGISTER_USER_KERNEL("cublas_fused_mlp_grad")             \
      .SetCreateFn<CublasFusedMLPGradKernel<dtype>>()         \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value)) \
        .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                               \
            const int64_t weight_size = ctx->input_size("weight"); \
            const Shape& dy_shape = ctx->InputShape("dy", 0); \
            int64_t m = dy_shape.At(0); \
            int64_t k = dy_shape.At(1); \
            int64_t tmp_buffer_size = 0; \
            for(int idx = weight_size-1; idx > 0; idx--){ \
              const Shape& weight_shape = ctx->InputShape("weight", idx); \
              k = weight_shape.At(1); \
              printf("M is: %ld, K is: %ld \n", m, k); \
              tmp_buffer_size += GetCudaAlignedSize(m * k * sizeof(dtype)); \
            } \
            printf("Success infer tmp \n"); \
            return tmp_buffer_size;                                                         \
        });

REGISTER_CUBLAS_FUSED_MLP_GRAD_KERNEL(float)
REGISTER_CUBLAS_FUSED_MLP_GRAD_KERNEL(double)
REGISTER_CUBLAS_FUSED_MLP_GRAD_KERNEL(half)

}  // namespace

}  // namespace oneflow

#endif  // CUDA_VERSION >= 11060
