#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/kernel_util.cuh"

#include <float.h>

namespace oneflow {

namespace {

// NOTE(Liang Depeng): refer to
// https://stackoverflow.com/questions/17371275/implementing-max-reduce-in-cuda
template<typename T>
__global__ void ReduceMaxMinPerLayer(const T *input_ptr, const int64_t elements, T *max_ptr,
                                     T *min_ptr) {
  extern __shared__ unsigned char shared_max_min_memory[];
  T *shared_max = reinterpret_cast<T *>(shared_max_min_memory);
  T *shared_min = shared_max + blockDim.x;

  int64_t tid = threadIdx.x;
  int64_t gid = (blockDim.x * blockIdx.x) + tid;
  shared_max[tid] = -FLT_MAX;
  shared_min[tid] = -FLT_MAX;

  while (gid < elements) {
    shared_max[tid] = max(shared_max[tid], input_ptr[gid]);
    shared_min[tid] = max(shared_min[tid], -input_ptr[gid]);
    gid += gridDim.x * blockDim.x;
  }
  __syncthreads();
  gid = (blockDim.x * blockIdx.x) + tid;
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s && gid < elements) {
      shared_max[tid] = max(shared_max[tid], shared_max[tid + s]);
      shared_min[tid] = max(shared_min[tid], shared_min[tid + s]);
    }
    __syncthreads();
  }

  if (tid == 0) {
    gpu_atomic_max(max_ptr, shared_max[0]);
    gpu_atomic_max(min_ptr, shared_min[0]);
  }
}

template<typename T>
__global__ void ReduceMaxMinPerChannel(const T *input_ptr, const int64_t elements,
                                       const int64_t num_channels, const int64_t panel_size,
                                       T *max_ptr, T *min_ptr) {
  extern __shared__ unsigned char shared_max_min_memory[];
  T *shared_max = reinterpret_cast<T *>(shared_max_min_memory);
  T *shared_min = shared_max + blockDim.x;

  int64_t cur_channel = blockIdx.x;
  int64_t tid = threadIdx.x;

  while (cur_channel < num_channels) {
    shared_max[tid] = -FLT_MAX;
    shared_min[tid] = -FLT_MAX;

    int64_t index = (panel_size * cur_channel) + tid;
    int64_t end = panel_size * (cur_channel + 1);

    while (index < end && index < elements) {
      shared_max[tid] = max(shared_max[tid], input_ptr[index]);
      shared_min[tid] = max(shared_min[tid], -input_ptr[index]);
      index += blockDim.x;
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
      if (tid < s) {
        shared_max[tid] = max(shared_max[tid], shared_max[tid + s]);
        shared_min[tid] = max(shared_min[tid], shared_min[tid + s]);
      }
      __syncthreads();
    }

    if (tid == 0) {
      gpu_atomic_max(&max_ptr[cur_channel], shared_max[0]);
      gpu_atomic_max(&min_ptr[cur_channel], shared_min[0]);
    }

    __syncthreads();
    cur_channel += gridDim.x;
  }
}

template<typename T>
__global__ void InitMaxMin(const int64_t elements, T *max_ptr, T *min_ptr) {
  int64_t tid = threadIdx.x;
  int64_t gid = (blockDim.x * blockIdx.x) + tid;

  while (gid < elements) {
    max_ptr[gid] = -FLT_MAX;
    min_ptr[gid] = -FLT_MAX;
    gid += gridDim.x * blockDim.x;
  }
}

template<typename T>
__global__ void CalScaleZeroPointSymmetric(const T *max_ptr, const T *min_ptr,
                                           const int64_t elements, const double quantize_to_bit,
                                           T *scale, T *zero_point) {
  int64_t tid = threadIdx.x;
  int64_t gid = (blockDim.x * blockIdx.x) + tid;

  while (gid < elements) {
    T weight_max = max(fabs(max_ptr[gid]), fabs(min_ptr[gid]));
    T denominator = static_cast<T>(pow(2.0, quantize_to_bit - 1)) - 1;
    scale[gid] = weight_max / denominator;
    zero_point[gid] = 0;
    gid += gridDim.x * blockDim.x;
  }
}

template<typename T>
__global__ void CalScaleZeroPointAffine(const T *max_ptr, const T *min_ptr, const int64_t elements,
                                        const double quantize_to_bit, T *scale, T *zero_point) {
  int64_t tid = threadIdx.x;
  int64_t gid = (blockDim.x * blockIdx.x) + tid;

  while (gid < elements) {
    T denominator = static_cast<T>(pow(2.0, quantize_to_bit)) - 1;
    T min = -min_ptr[gid];
    T s = (max_ptr[gid] - min) / denominator;
    scale[gid] = s;
    zero_point[gid] = -min / s;
    gid += gridDim.x * blockDim.x;
  }
}

}  // namespace

#define LAUNCH_CUDA_KERNEL(func, device_ctx_ptr, thread_num, shared_mem_size, ...)     \
  func<<<SMBlocksNum4ThreadsNum(thread_num), kCudaThreadsNumPerBlock, shared_mem_size, \
         (device_ctx_ptr)->hip_stream()>>>(__VA_ARGS__)

template<typename T>
class GpuMinMaxObserverKernel final : public user_op::OpKernel {
 public:
  GpuMinMaxObserverKernel() = default;
  ~GpuMinMaxObserverKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext *ctx) const override {
    const user_op::Tensor *in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor *scale = ctx->Tensor4ArgNameAndIndex("scale", 0);
    user_op::Tensor *zero_point = ctx->Tensor4ArgNameAndIndex("zero_point", 0);
    user_op::Tensor *tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    const std::string quantize_scheme = ctx->Attr<std::string>("quantize_scheme");
    const int32_t quantize_to_bit = ctx->Attr<int32_t>("quantize_to_bit");
    const bool per_layer_quantize = ctx->Attr<bool>("per_layer_quantize");

    const int64_t elements = in->shape().elem_cnt();
    const int64_t channel = scale->shape().At(0);
    const int64_t panel_size = elements / channel;
    T *max_ptr = tmp_buffer->mut_dptr<T>();
    T *min_ptr = max_ptr + channel;

    LAUNCH_CUDA_KERNEL((InitMaxMin<T>), ctx->device_ctx(), channel, 0, channel, max_ptr, min_ptr);

    if (per_layer_quantize) {
      LAUNCH_CUDA_KERNEL((ReduceMaxMinPerLayer<T>), ctx->device_ctx(), elements,
                         kCudaThreadsNumPerBlock * 2 * sizeof(T), in->dptr<T>(), elements, max_ptr,
                         min_ptr);
    } else {  // per-channel quantize
      // NOTE(Liang Depeng): each block of threads will be responsible for
      //                     computing the max and min values of the whole channel.
      LAUNCH_CUDA_KERNEL((ReduceMaxMinPerChannel<T>), ctx->device_ctx(),
                         channel * kCudaThreadsNumPerBlock, kCudaThreadsNumPerBlock * 2 * sizeof(T),
                         in->dptr<T>(), elements, channel, panel_size, max_ptr, min_ptr);
    }

    if (quantize_scheme == "symmetric") {
      LAUNCH_CUDA_KERNEL((CalScaleZeroPointSymmetric<T>), ctx->device_ctx(), channel, 0, max_ptr,
                         min_ptr, channel, static_cast<double>(quantize_to_bit),
                         scale->mut_dptr<T>(), zero_point->mut_dptr<T>());
    } else {  // quantize_scheme == "affine"
      LAUNCH_CUDA_KERNEL((CalScaleZeroPointAffine<T>), ctx->device_ctx(), channel, 0, max_ptr,
                         min_ptr, channel, static_cast<double>(quantize_to_bit),
                         scale->mut_dptr<T>(), zero_point->mut_dptr<T>());
    }
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_MIN_MAX_OBSERVER_KERNEL(dtype)                                        \
  REGISTER_USER_KERNEL("min_max_observer")                                             \
      .SetCreateFn<GpuMinMaxObserverKernel<dtype>>()                                   \
      .SetIsMatchedHob((user_op::HobDeviceTag() == DeviceType::kGPU)                   \
                       & (user_op::HobDataType("in", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn([](user_op::InferContext *ctx) -> size_t {                    \
        size_t tmp_buffer_size = 1;                                                    \
        if (ctx->Attr<bool>("per_layer_quantize") == false) {                          \
          const Shape *in_shape = ctx->Shape4ArgNameAndIndex("in", 0);                 \
          tmp_buffer_size = in_shape->At(0);                                           \
        }                                                                              \
        return 2 * tmp_buffer_size * sizeof(dtype);                                    \
      })

REGISTER_MIN_MAX_OBSERVER_KERNEL(float);
REGISTER_MIN_MAX_OBSERVER_KERNEL(double);

}  // namespace oneflow
