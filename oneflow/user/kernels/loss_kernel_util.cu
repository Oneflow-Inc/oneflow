#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/user/kernels/loss_kernel_util.h"

namespace oneflow {
namespace user_op {
namespace loss {

template<typename T>
__global__ void ApplyLossReductionImplKernel(int64_t elem_cnt, const T* tmp_out, T* out,
                                             bool is_reduce_mean) {
  __shared__ T outs[kCudaThreadsNumPerBlock];
  outs[threadIdx.x] = static_cast<T>(0);
  for (int i = threadIdx.x; i < elem_cnt; i += kCudaThreadsNumPerBlock) {
    outs[threadIdx.x] += tmp_out[i];
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    *out = static_cast<T>(0);
    for (int i = 0; i < kCudaThreadsNumPerBlock; ++i) { *out += outs[i]; }
    if (is_reduce_mean) { *out /= elem_cnt; }
  }
}
template<>
__global__ void ApplyLossReductionImplKernel<half>(int64_t elem_cnt, const half* tmp_out, half* out,
                                                   bool is_reduce_mean) {
  __shared__ half outs[kCudaThreadsNumPerBlock];
  outs[threadIdx.x] = __float2half(0.0);
  for (int i = threadIdx.x; i < elem_cnt; i += kCudaThreadsNumPerBlock) {
    outs[threadIdx.x] = __hadd(outs[threadIdx.x], tmp_out[i]);
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    *out = __float2half(0.0);
    for (int i = 0; i < kCudaThreadsNumPerBlock; ++i) { *out = __hadd(*out, outs[i]); }
    if (is_reduce_mean) { *out = __float2half(__half2float(*out) / elem_cnt); }
  }
}

template<typename T>
void ApplyLossReduction(DeviceCtx* ctx, int64_t elem_cnt, const T* tmp_out, T* out,
                        const ReductionType reduction_type) {
  if ((reduction_type != ReductionType::kMean) && (reduction_type != ReductionType::kSum)) {
    UNIMPLEMENTED();
    return;
  }
  ApplyLossReductionImplKernel<<<1, kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      elem_cnt, tmp_out, out, reduction_type == ReductionType::kMean);
}
template<>
void ApplyLossReduction<float16>(DeviceCtx* ctx, int64_t elem_cnt, const float16* tmp_out,
                                 float16* out, const ReductionType reduction_type) {
  if ((reduction_type != ReductionType::kMean) && (reduction_type != ReductionType::kSum)) {
    UNIMPLEMENTED();
    return;
  }
  ApplyLossReductionImplKernel<<<1, kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      elem_cnt, reinterpret_cast<const half*>(tmp_out), reinterpret_cast<half*>(out),
      reduction_type == ReductionType::kMean);
}
#define SPECIALIZE_APPLY_LOSS_REDUCTION(dtype)                                                     \
  template void ApplyLossReduction<dtype>(DeviceCtx * ctx, int64_t elem_cnt, const dtype* tmp_out, \
                                          dtype* out, const ReductionType reduction_type);

SPECIALIZE_APPLY_LOSS_REDUCTION(float)
SPECIALIZE_APPLY_LOSS_REDUCTION(double)
SPECIALIZE_APPLY_LOSS_REDUCTION(float16)

}  // namespace loss
}  // namespace user_op
}  // namespace oneflow
