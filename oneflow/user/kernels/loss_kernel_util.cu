#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <hipcub/hipcub.hpp>
#include "oneflow/user/kernels/loss_kernel_util.h"

namespace oneflow {
namespace user_op {
namespace loss {

template<typename T>
__global__ void ApplyLossReductionImpl(int64_t elem_cnt, double inv_elem_cnt, const T* tmp_out,
                                       T* out, bool is_reduce_mean) {
  typedef hipcub::BlockReduce<T, kCudaThreadsNumPerBlock> BlockReduce;
  __shared__ typename BlockReduce::TempStorage cub_reduce_tmp_storage;
  T thread_sum = static_cast<T>(0);
  for (int i = threadIdx.x; i < elem_cnt; i += kCudaThreadsNumPerBlock) {
    thread_sum += tmp_out[i];
  }
  __syncthreads();
  T block_sum = BlockReduce(cub_reduce_tmp_storage).Reduce(thread_sum, hipcub::Sum());
  if (threadIdx.x == 0) {
    T out_val = block_sum;
    if (is_reduce_mean) { out_val *= static_cast<T>(inv_elem_cnt); }
    *out = out_val;
  }
}

template<>
__global__ void ApplyLossReductionImpl<half>(int64_t elem_cnt, double inv_elem_cnt,
                                             const half* tmp_out, half* out, bool is_reduce_mean) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  typedef hipcub::BlockReduce<half, kCudaThreadsNumPerBlock> BlockReduce;
  __shared__ typename BlockReduce::TempStorage cub_reduce_tmp_storage;
  half thread_sum = __float2half(0.0);
  for (int i = threadIdx.x; i < elem_cnt; i += kCudaThreadsNumPerBlock) {
    thread_sum = __hadd(thread_sum, tmp_out[i]);
  }
  __syncthreads();
  half block_sum = BlockReduce(cub_reduce_tmp_storage).Reduce(thread_sum, hipcub::Sum());
  if (threadIdx.x == 0) {
    half out_val = block_sum;
    if (is_reduce_mean) {
      out_val = __float2half(__half2float(*out) * static_cast<float>(inv_elem_cnt));
    }
    *out = out_val;
  }
#else
  printf("use half need nvcc arch >= 530");
  assert(false);
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)*/
}

template<DeviceType device_type, typename T>
RETURN_VOID_IF_GPU(device_type)
ApplyLossReductionIfNeed(DeviceCtx* ctx, int64_t elem_cnt, const T* tmp_out, T* out,
                         const ReductionType reduction_type) {
  if (reduction_type == ReductionType::kNone) { return; }
  if ((reduction_type != ReductionType::kMean) && (reduction_type != ReductionType::kSum)) {
    UNIMPLEMENTED();
    return;
  }
  ApplyLossReductionImpl<<<1, kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      elem_cnt, static_cast<double>(1.0 / elem_cnt), tmp_out, out,
      reduction_type == ReductionType::kMean);
}

#define SPECIALIZE_APPLY_LOSS_REDUCTION(device_type, dtype)                              \
  template RETURN_VOID_IF_GPU(device_type) ApplyLossReductionIfNeed<device_type, dtype>( \
      DeviceCtx * ctx, int64_t elem_cnt, const dtype* tmp_out, dtype* out,               \
      const ReductionType reduction_type);

SPECIALIZE_APPLY_LOSS_REDUCTION(DeviceType::kGPU, half)
SPECIALIZE_APPLY_LOSS_REDUCTION(DeviceType::kGPU, float)
SPECIALIZE_APPLY_LOSS_REDUCTION(DeviceType::kGPU, double)

}  // namespace loss
}  // namespace user_op
}  // namespace oneflow
