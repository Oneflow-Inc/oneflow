/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/where_kernel_util.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

template<typename T, typename CondT>
struct WhereFunctor {
  OF_DEVICE_FUNC T operator()(CondT cond, T lhs, T rhs) const {
    return static_cast<bool>(cond) ? lhs : rhs;
  }
};

template<typename T, typename CondT>
struct WhereScalarXFunctor {
  OF_DEVICE_FUNC explicit WhereScalarXFunctor(T scalar) : x_scalar(scalar) {}
  OF_DEVICE_FUNC T operator()(CondT cond, T rhs) const {
    return static_cast<bool>(cond) ? x_scalar : rhs;
  }
  const T x_scalar;
};

template<typename T, typename CondT>
struct WhereScalarYFunctor {
  OF_DEVICE_FUNC explicit WhereScalarYFunctor(T scalar) : y_scalar(scalar) {}
  OF_DEVICE_FUNC T operator()(CondT cond, T lhs) const {
    return static_cast<bool>(cond) ? lhs : y_scalar;
  }
  const T y_scalar;
};

template<typename T, typename CondT>
struct WhereScalarXYFunctor {
  OF_DEVICE_FUNC explicit WhereScalarXYFunctor(T x_scalar, T y_scalar)
      : x_scalar(x_scalar), y_scalar(y_scalar) {}
  OF_DEVICE_FUNC T operator()(CondT cond) const {
    return static_cast<bool>(cond) ? x_scalar : y_scalar;
  }
  const T x_scalar;
  const T y_scalar;
};

}  // namespace

template<typename T, typename CondT>
struct WhereKernelUtil<DeviceType::kCUDA, T, CondT> {
  static void Where(ep::Stream* stream, const int64_t elem_cnt, const CondT* cond, const T* lhs,
                    const T* rhs, T* out) {
    cuda::elementwise::Ternary(WhereFunctor<T, CondT>(), elem_cnt, out, cond, lhs, rhs,
                               stream->As<ep::CudaStream>()->hip_stream());
  }
  static void WhereXScalar(ep::Stream* stream, const int64_t elem_cnt, const CondT* cond,
                           const T x_scalar, const T* rhs, T* out) {
    cuda::elementwise::Binary(WhereScalarXFunctor<T, CondT>(x_scalar), elem_cnt, out, cond, rhs,
                              stream->As<ep::CudaStream>()->hip_stream());
  }
  static void WhereYScalar(ep::Stream* stream, const int64_t elem_cnt, const CondT* cond,
                           const T* lhs, const T y_scalar, T* out) {
    cuda::elementwise::Binary(WhereScalarYFunctor<T, CondT>(y_scalar), elem_cnt, out, cond, lhs,
                              stream->As<ep::CudaStream>()->hip_stream());
  }
  static void WhereXYScalar(ep::Stream* stream, const int64_t elem_cnt, const CondT* cond,
                            const T x_scalar, const T y_scalar, T* out) {
    cuda::elementwise::Unary(WhereScalarXYFunctor<T, CondT>(x_scalar, y_scalar), elem_cnt, out,
                             cond, stream->As<ep::CudaStream>()->hip_stream());
  }
};

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_WHERE_FUNCTOR, (DeviceType::kCUDA),
                                 ARITHMETIC_DATA_TYPE_SEQ FLOAT16_DATA_TYPE_SEQ BOOL_DATA_TYPE_SEQ,
                                 INT_DATA_TYPE_SEQ BOOL_DATA_TYPE_SEQ)

}  // namespace oneflow
