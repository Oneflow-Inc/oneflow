#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ndarray/ndarray_util.h"
#include "oneflow/core/cuda/atomic.cuh"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/ep/include/primitive/fill.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/layer_norm.cuh"

namespace oneflow {

namespace {

template<typename SRC, typename DST, bool do_scale, bool do_center>
struct AffineStore {
  AffineStore(DST* normalized, DST* y, int64_t row_size, const DST* gamma, const DST* beta)
      : normalized(normalized), y(y), row_size(row_size), gamma(gamma), beta(beta) {}
  template<int N>
  __device__ void store(const SRC* src, int64_t row, int64_t col) {
    cuda::layer_norm::Pack<DST, N> y_pack;
    cuda::layer_norm::Pack<DST, N> normalized_pack;
    cuda::layer_norm::Pack<DST, N> gamma_pack;
    cuda::layer_norm::Pack<DST, N> beta_pack;
    const int64_t offset = (row * row_size + col) / N;
    const int64_t gamma_offset = col / N;
    if (do_scale) {
      gamma_pack.storage =
          *(reinterpret_cast<const cuda::layer_norm::PackType<DST, N>*>(gamma) + gamma_offset);
    } else {
#pragma unroll
      for (int i = 0; i < N; ++i) { gamma_pack.elem[i] = 1; }
    }
    if (do_center) {
      beta_pack.storage =
          *(reinterpret_cast<const cuda::layer_norm::PackType<DST, N>*>(beta) + gamma_offset);
    } else {
#pragma unroll
      for (int i = 0; i < N; ++i) { beta_pack.elem[i] = 0; }
    }
#pragma unroll
    for (int i = 0; i < N; ++i) {
      DST normalized_i = static_cast<DST>(src[i]);
      if (do_scale) { normalized_pack.elem[i] = normalized_i; }
      if (do_scale || do_center) {
        y_pack.elem[i] = normalized_i * gamma_pack.elem[i] + beta_pack.elem[i];
      } else {
        y_pack.elem[i] = normalized_i;
      }
    }
    *(reinterpret_cast<cuda::layer_norm::PackType<DST, N>*>(y) + offset) = y_pack.storage;
    if (do_scale) {
      *(reinterpret_cast<cuda::layer_norm::PackType<DST, N>*>(normalized) + offset) =
          normalized_pack.storage;
    }
  }
  DST* normalized;
  DST* y;
  int64_t row_size;
  const DST* gamma;
  const DST* beta;
};

template<typename SRC, typename DST, bool do_scale>
struct ScaleLoad {
  ScaleLoad(const SRC* src, const SRC* gamma, int64_t row_size)
      : src(src), gamma(gamma), row_size(row_size) {}
  template<int N>
  __device__ void load(DST* dst, int64_t row, int64_t col) const {
    cuda::layer_norm::Pack<SRC, N> src_pack;
    cuda::layer_norm::Pack<SRC, N> gamma_pack;
    const int64_t offset = (row * row_size + col) / N;
    const int64_t gamma_offset = col / N;
    src_pack.storage = *(reinterpret_cast<const cuda::layer_norm::PackType<SRC, N>*>(src) + offset);
    if (do_scale) {
      gamma_pack.storage =
          *(reinterpret_cast<const cuda::layer_norm::PackType<SRC, N>*>(gamma) + gamma_offset);
    } else {
#pragma unroll
      for (int i = 0; i < N; ++i) { gamma_pack.elem[i] = static_cast<SRC>(1); }
    }
#pragma unroll
    for (int i = 0; i < N; ++i) {
      dst[i] = static_cast<DST>(src_pack.elem[i] * gamma_pack.elem[i]);
    }
  }
  const SRC* src;
  const SRC* gamma;
  int64_t row_size;
};

template<typename SRC, typename DST, bool do_add>
struct AddStore {
  AddStore(const DST* add_to_output, DST* dst, int64_t row_size)
      : add_to_output(add_to_output), dst(dst), row_size(row_size) {}
  template<int N>
  __device__ void store(const SRC* src, int64_t row, int64_t col) {
    cuda::layer_norm::Pack<DST, N> add_to_output_pack;
    cuda::layer_norm::Pack<DST, N> dst_pack;
    const int64_t offset = (row * row_size + col) / N;
    if (do_add) {
      add_to_output_pack.storage =
          *(reinterpret_cast<const cuda::layer_norm::PackType<DST, N>*>(add_to_output) + offset);
    }
#pragma unroll
    for (int i = 0; i < N; ++i) {
      if (do_add) {
        dst_pack.elem[i] = static_cast<DST>(src[i]) + add_to_output_pack.elem[i];
      } else {
        dst_pack.elem[i] = static_cast<DST>(src[i]);
      }
    }
    *(reinterpret_cast<cuda::layer_norm::PackType<DST, N>*>(dst) + offset) = dst_pack.storage;
  }
  const DST* add_to_output;
  DST* dst;
  int64_t row_size;
};

constexpr int64_t kLayerNormParamGradGpuBlockSize = 512;

int64_t GetLayerNormParamGradBlockSize() { return kLayerNormParamGradGpuBlockSize; }

int64_t GetLayerNormParamGradNumBlocks(const int64_t elem_cnt) {
  return std::min(static_cast<int>((elem_cnt + kLayerNormParamGradGpuBlockSize - 1)
                                   / kLayerNormParamGradGpuBlockSize),
                  256);
}

template<typename T>
int64_t GetParamGradDynamicSharedMemorySize(const int64_t instance_size) {
  return 2 * instance_size * sizeof(T);
}

template<>
int64_t GetParamGradDynamicSharedMemorySize<float16>(const int64_t instance_size) {
  return 2 * instance_size * sizeof(float);
}

template<typename T, typename I>
__global__ void LayerNormParamGradImpl(const I n, const I instance_size, const T* dy,
                                       const T* normalized, const T* gamma, T* gamma_diff,
                                       T* beta_diff, T* normalized_diff) {
  extern __shared__ __align__(sizeof(double)) unsigned char bw_shared_buf[];
  auto* gamma_diff_sum_buf = reinterpret_cast<T*>(bw_shared_buf);
  auto* beta_diff_sum_buf = gamma_diff_sum_buf + instance_size;
  const I tid = threadIdx.x;
  for (I elem_id = tid; elem_id < instance_size; elem_id += blockDim.x) {
    gamma_diff_sum_buf[elem_id] = 0;
    beta_diff_sum_buf[elem_id] = 0;
  }
  __syncthreads();
  CUDA_1D_KERNEL_LOOP_T(I, i, n) {
    const I elem_id = i % instance_size;
    T dy_val = dy[i];
    T normalized_val = normalized[i];
    cuda::atomic::Add(&gamma_diff_sum_buf[elem_id], dy_val * normalized_val);
    cuda::atomic::Add(&beta_diff_sum_buf[elem_id], dy_val);
    T gamma_val = gamma[elem_id];
    normalized_diff[i] = gamma_val * dy_val;
  }
  __syncthreads();
  for (I elem_id = tid; elem_id < instance_size; elem_id += blockDim.x) {
    cuda::atomic::Add(gamma_diff + elem_id, gamma_diff_sum_buf[elem_id]);
    cuda::atomic::Add(beta_diff + elem_id, beta_diff_sum_buf[elem_id]);
  }
}

template<typename I>
__global__ void LayerNormParamGradHalfImpl(const I n, const I instance_size, const half* dy,
                                           const half* normalized, const half* gamma,
                                           half* tmp_gamma_diff, half* tmp_beta_diff,
                                           half* normalized_diff) {
  extern __shared__ __align__(sizeof(double)) unsigned char bw_shared_buf[];
  auto* gamma_diff_sum_buf = reinterpret_cast<float*>(bw_shared_buf);
  auto* beta_diff_sum_buf = gamma_diff_sum_buf + instance_size;
  const I tid = threadIdx.x;
  for (I elem_id = tid; elem_id < instance_size; elem_id += blockDim.x) {
    gamma_diff_sum_buf[elem_id] = 0;
    beta_diff_sum_buf[elem_id] = 0;
  }
  __syncthreads();
  CUDA_1D_KERNEL_LOOP_T(I, i, n) {
    const I elem_id = i % instance_size;
    half dy_val = dy[i];
    half normalized_val = normalized[i];
    cuda::atomic::Add(&gamma_diff_sum_buf[elem_id],
                      __half2float(dy_val) * __half2float(normalized_val));
    cuda::atomic::Add(&beta_diff_sum_buf[elem_id], __half2float(dy_val));
    half gamma_val = gamma[elem_id];
    normalized_diff[i] = __hmul(gamma_val, dy_val);
  }
  __syncthreads();
  for (I elem_id = tid; elem_id < instance_size; elem_id += blockDim.x) {
    const I offset = blockIdx.x * instance_size + elem_id;
    tmp_gamma_diff[offset] = __float2half(gamma_diff_sum_buf[elem_id]);
    tmp_beta_diff[offset] = __float2half(beta_diff_sum_buf[elem_id]);
  }
}

template<typename T, bool do_scale, bool do_center>
void LayerNormForwardGpu(ep::Stream* stream, const int64_t num_instances, const int64_t norm_size,
                         const double epsilon, const T* x_ptr, const T* gamma_ptr,
                         const T* beta_ptr, T* normalized_ptr, T* y_ptr, user_op::Tensor* mean,
                         user_op::Tensor* inv_variance) {
  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  cuda::layer_norm::DirectLoad<T, ComputeType> load(x_ptr, norm_size);
  AffineStore<ComputeType, T, do_scale, do_center> store(normalized_ptr, y_ptr, norm_size,
                                                         gamma_ptr, beta_ptr);
  cuda::layer_norm::DispatchLayerNorm<decltype(load), decltype(store), ComputeType>(
      stream->As<ep::CudaStream>()->hip_stream(), load, store, num_instances, norm_size, epsilon,
      mean->mut_dptr<ComputeType>(), inv_variance->mut_dptr<ComputeType>());
}

template<typename T>
void DispatchLayerNormForwardGpu(ep::Stream* stream, const int64_t num_instances,
                                 const int64_t norm_size, const double epsilon, const T* x_ptr,
                                 const T* gamma_ptr, const T* beta_ptr, T* normalized_ptr, T* y_ptr,
                                 user_op::Tensor* mean, user_op::Tensor* inv_variance) {
  if (gamma_ptr != nullptr && beta_ptr != nullptr) {
    LayerNormForwardGpu<T, true, true>(stream, num_instances, norm_size, epsilon, x_ptr, gamma_ptr,
                                       beta_ptr, normalized_ptr, y_ptr, mean, inv_variance);
  } else if (gamma_ptr != nullptr && beta_ptr == nullptr) {
    LayerNormForwardGpu<T, true, false>(stream, num_instances, norm_size, epsilon, x_ptr, gamma_ptr,
                                        beta_ptr, normalized_ptr, y_ptr, mean, inv_variance);
  } else if (gamma_ptr == nullptr && beta_ptr != nullptr) {
    LayerNormForwardGpu<T, false, true>(stream, num_instances, norm_size, epsilon, x_ptr, gamma_ptr,
                                        beta_ptr, normalized_ptr, y_ptr, mean, inv_variance);
  } else {
    LayerNormForwardGpu<T, false, false>(stream, num_instances, norm_size, epsilon, x_ptr,
                                         gamma_ptr, beta_ptr, normalized_ptr, y_ptr, mean,
                                         inv_variance);
  }
}

template<typename T, bool do_scale, bool do_add>
void LayerNormBackwardGpu(ep::Stream* stream, const int64_t num_instances, const int64_t norm_size,
                          const T* dy_ptr, const T* x_ptr, const user_op::Tensor* mean,
                          const user_op::Tensor* inv_variance, const T* gamma_ptr,
                          const T* add_to_output_ptr, T* dx_ptr) {
  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  cuda::layer_norm::DirectLoad<T, ComputeType> load_x(x_ptr, norm_size);
  ScaleLoad<T, ComputeType, do_scale> load_scaled_dy(dy_ptr, gamma_ptr, norm_size);
  AddStore<ComputeType, T, do_add> store(add_to_output_ptr, dx_ptr, norm_size);
  OF_CUDA_CHECK((cuda::layer_norm::DispatchLayerNormGrad<decltype(load_x), decltype(load_scaled_dy),
                                                         decltype(store), ComputeType>(
      stream->As<ep::CudaStream>()->hip_stream(), load_x, load_scaled_dy, store,
      mean->dptr<ComputeType>(), inv_variance->dptr<ComputeType>(), num_instances, norm_size)));
}

template<typename T, bool do_scale>
void DispatchLayerNormBackwardDoAdd(ep::Stream* stream, const int64_t num_instances,
                                    const int64_t norm_size, const T* dy_ptr, const T* x_ptr,
                                    const user_op::Tensor* mean,
                                    const user_op::Tensor* inv_variance, const T* gamma_ptr,
                                    const T* add_to_output_ptr, T* dx_ptr) {
  if (add_to_output_ptr != nullptr) {
    LayerNormBackwardGpu<T, do_scale, true>(stream, num_instances, norm_size, dy_ptr, x_ptr, mean,
                                            inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
  } else {
    LayerNormBackwardGpu<T, do_scale, false>(stream, num_instances, norm_size, dy_ptr, x_ptr, mean,
                                             inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
  }
}

template<typename T>
void LaunchLayerNormBackward(ep::Stream* stream, const int64_t num_instances,
                             const int64_t norm_size, const T* dy_ptr, const T* x_ptr,
                             const user_op::Tensor* mean, const user_op::Tensor* inv_variance,
                             const T* gamma_ptr, const T* add_to_output_ptr, T* dx_ptr) {
  if (gamma_ptr != nullptr) {
    DispatchLayerNormBackwardDoAdd<T, true>(stream, num_instances, norm_size, dy_ptr, x_ptr, mean,
                                            inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
  } else {
    DispatchLayerNormBackwardDoAdd<T, false>(stream, num_instances, norm_size, dy_ptr, x_ptr, mean,
                                             inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
  }
}

}  // namespace

template<typename T>
class LayerNormGpuKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  LayerNormGpuKernel() = default;
  ~LayerNormGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    user_op::Tensor* normalized =
        ctx->has_input("gamma", 0) ? ctx->Tensor4ArgNameAndIndex("normalized", 0) : y;
    const double epsilon = ctx->Attr<double>("epsilon");
    CHECK_GE(epsilon, HIPDNN_BN_MIN_EPSILON);
    const int64_t num_instances = mean->shape().elem_cnt();
    const int64_t norm_size = x->shape().elem_cnt() / num_instances;
    const T* gamma_ptr = nullptr;
    const T* beta_ptr = nullptr;
    if (ctx->has_input("gamma", 0)) {
      const user_op::Tensor* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
      gamma_ptr = gamma->dptr<T>();
      CHECK_EQ(gamma->shape().elem_cnt(), norm_size);
    }
    if (ctx->has_input("beta", 0)) { beta_ptr = ctx->Tensor4ArgNameAndIndex("beta", 0)->dptr<T>(); }
    DispatchLayerNormForwardGpu<T>(ctx->stream(), num_instances, norm_size, epsilon, x->dptr<T>(),
                                   gamma_ptr, beta_ptr, normalized->mut_dptr<T>(), y->mut_dptr<T>(),
                                   mean, inv_variance);
  };
};

#define REGISTER_LAYER_NORM_CUDA_KERNEL(dtype)                         \
  REGISTER_USER_KERNEL("layer_norm")                                   \
      .SetCreateFn<LayerNormGpuKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value));

REGISTER_LAYER_NORM_CUDA_KERNEL(float)
REGISTER_LAYER_NORM_CUDA_KERNEL(double)
REGISTER_LAYER_NORM_CUDA_KERNEL(half)

template<typename T>
class LayerNormGradGpuKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  LayerNormGradGpuKernel() = default;
  ~LayerNormGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const int64_t num_instances = mean->shape().elem_cnt();
    const int64_t norm_size = x->shape().elem_cnt() / num_instances;
    const T* gamma_ptr = nullptr;
    if (ctx->has_input("gamma", 0)) {
      gamma_ptr = ctx->Tensor4ArgNameAndIndex("gamma", 0)->dptr<T>();
    }
    const T* add_to_output_ptr = nullptr;
    if (ctx->has_input("_add_to_output", 0)) {
      const user_op::Tensor* add_to_output = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      CHECK_EQ(add_to_output->data_type(), dx->data_type());
      CHECK_EQ(add_to_output->shape(), dx->shape());
      add_to_output_ptr = add_to_output->dptr<T>();
    }
    LaunchLayerNormBackward<T>(ctx->stream(), num_instances, norm_size, dy->dptr<T>(), x->dptr<T>(),
                               mean, inv_variance, gamma_ptr, add_to_output_ptr, dx->mut_dptr<T>());
  };
};

#define REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(dtype)                                        \
  REGISTER_USER_KERNEL("layer_norm_grad")                                                  \
      .SetCreateFn<LayerNormGradGpuKernel<dtype>>()                                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                     \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value))    \
      .SetInplaceProposalFn(                                                               \
          [](const user_op::InferContext& ctx,                                             \
             const user_op::AddInplaceArgPair& AddInplaceArgPairFn) -> Maybe<void> {       \
            if (ctx.has_input("_add_to_output", 0)) {                                      \
              OF_RETURN_IF_ERROR(AddInplaceArgPairFn("dx", 0, "_add_to_output", 0, true)); \
            }                                                                              \
            return Maybe<void>::Ok();                                                      \
          });

REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(float)
REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(double)
REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(half)

template<typename T>
class LayerNormParamGradGpuKernel final : public user_op::OpKernel,
                                          public user_op::CudaGraphSupport {
 public:
  LayerNormParamGradGpuKernel() = default;
  ~LayerNormParamGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    using NdUtil = NdarrayUtil<DeviceType::kCUDA, T>;
    auto Val = NdUtil::GetValNdarrayBuilder();
    auto Var = NdUtil::GetVarNdarrayBuilder();
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* beta_diff = ctx->Tensor4ArgNameAndIndex("beta_diff", 0);
    user_op::Tensor* gamma_diff = ctx->Tensor4ArgNameAndIndex("gamma_diff", 0);
    user_op::Tensor* normalized_diff = ctx->Tensor4ArgNameAndIndex("normalized_diff", 0);
    user_op::Tensor* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const bool has_beta_diff = beta_diff != nullptr;
    const bool has_gamma_diff = gamma_diff != nullptr;
    const bool has_normalized_diff = normalized_diff != nullptr;
    const bool has_gamma = gamma != nullptr;
    const int64_t begin_params_axis = ctx->Attr<int64_t>("begin_params_axis");
    const int64_t elem_cnt = dy->shape().elem_cnt();
    const int64_t m = dy->shape().Count(begin_params_axis);
    int max_active_blocks = 0;
    OF_CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks, LayerNormParamGradImpl<T, int64_t>, GetLayerNormParamGradBlockSize(),
        GetParamGradDynamicSharedMemorySize<T>(m)));
    if (has_gamma_diff && has_beta_diff && has_normalized_diff && max_active_blocks > 0) {
      const user_op::Tensor* normalized = ctx->Tensor4ArgNameAndIndex("normalized", 0);
      Memset<DeviceType::kCUDA>(ctx->stream(), gamma_diff->mut_dptr<T>(), 0,
                                gamma_diff->shape().elem_cnt() * sizeof(T));
      Memset<DeviceType::kCUDA>(ctx->stream(), beta_diff->mut_dptr<T>(), 0,
                                beta_diff->shape().elem_cnt() * sizeof(T));
      if (elem_cnt > static_cast<int64_t>(GetMaxVal<int32_t>() / 2)) {
        LayerNormParamGradImpl<T, int64_t>
            <<<GetLayerNormParamGradNumBlocks(elem_cnt), GetLayerNormParamGradBlockSize(),
               GetParamGradDynamicSharedMemorySize<T>(m),
               ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
                elem_cnt, m, dy->dptr<T>(), normalized->dptr<T>(), gamma->dptr<T>(),
                gamma_diff->mut_dptr<T>(), beta_diff->mut_dptr<T>(),
                normalized_diff->mut_dptr<T>());
      } else {
        LayerNormParamGradImpl<T, int32_t>
            <<<GetLayerNormParamGradNumBlocks(elem_cnt), GetLayerNormParamGradBlockSize(),
               GetParamGradDynamicSharedMemorySize<T>(m),
               ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
                static_cast<int32_t>(elem_cnt), static_cast<int32_t>(m), dy->dptr<T>(),
                normalized->dptr<T>(), gamma->dptr<T>(), gamma_diff->mut_dptr<T>(),
                beta_diff->mut_dptr<T>(), normalized_diff->mut_dptr<T>());
      }
    } else {
      if (has_beta_diff) {
        user_op::Tensor* reduce_buf = ctx->Tensor4ArgNameAndIndex("reduce_buf", 0);
        CHECK_EQ(m, beta_diff->shape().elem_cnt());
        CHECK_EQ(dy->shape().elem_cnt() % m, 0);
        const int64_t n = dy->shape().elem_cnt() / m;
        NdUtil::ReduceSum(ctx->stream(), Var({1, m}, beta_diff->mut_dptr<T>()),
                          Val({n, m}, dy->dptr<T>()), Var({n, m}, reduce_buf->mut_dptr<T>()));
      }
      if (has_gamma_diff) {
        const user_op::Tensor* normalized = ctx->Tensor4ArgNameAndIndex("normalized", 0);
        user_op::Tensor* reduce_buf = ctx->Tensor4ArgNameAndIndex("reduce_buf", 0);
        CHECK_EQ(m, gamma_diff->shape().elem_cnt());
        CHECK_EQ(dy->shape().elem_cnt() % m, 0);
        const int64_t n = dy->shape().elem_cnt() / m;
        NdUtil::BroadcastMul(ctx->stream(), Var({n, m}, reduce_buf->mut_dptr<T>()),
                             Val({n, m}, normalized->dptr<T>()), Val({n, m}, dy->dptr<T>()));
        NdUtil::ReduceSum(ctx->stream(), Var({1, m}, gamma_diff->mut_dptr<T>()),
                          Val({n, m}, reduce_buf->dptr<T>()),
                          Var({n, m}, reduce_buf->mut_dptr<T>()));
      }
      if (has_normalized_diff) {
        if (has_gamma) {
          CHECK_EQ(m, gamma->shape().elem_cnt());
          CHECK_EQ(dy->shape().elem_cnt() % m, 0);
          const int64_t n = dy->shape().elem_cnt() / m;
          NdUtil::BroadcastMul(ctx->stream(), Var({n, m}, normalized_diff->mut_dptr<T>()),
                               Val({n, m}, dy->dptr<T>()), Val({1, m}, gamma->dptr<T>()));
        } else {
          Memcpy<DeviceType::kCUDA>(ctx->stream(), normalized_diff->mut_dptr<void>(),
                                    dy->dptr<void>(),
                                    dy->shape().elem_cnt() * GetSizeOfDataType(dy->data_type()));
        }
      }
    }
  };
};

#define REGISTER_LAYER_NORM_PARAM_GRAD_CUDA_KERNEL(dtype)              \
  REGISTER_USER_KERNEL("layer_norm_param_grad")                        \
      .SetCreateFn<LayerNormParamGradGpuKernel<dtype>>()               \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value));

REGISTER_LAYER_NORM_PARAM_GRAD_CUDA_KERNEL(float)
REGISTER_LAYER_NORM_PARAM_GRAD_CUDA_KERNEL(double)

class LayerNormParamGradGpuHalfKernel final : public user_op::OpKernel,
                                              public user_op::CudaGraphSupport {
 public:
  LayerNormParamGradGpuHalfKernel() = default;
  ~LayerNormParamGradGpuHalfKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    using NdUtil = NdarrayUtil<DeviceType::kCUDA, float16>;
    auto Val = NdUtil::GetValNdarrayBuilder();
    auto Var = NdUtil::GetVarNdarrayBuilder();
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* beta_diff = ctx->Tensor4ArgNameAndIndex("beta_diff", 0);
    user_op::Tensor* gamma_diff = ctx->Tensor4ArgNameAndIndex("gamma_diff", 0);
    user_op::Tensor* normalized_diff = ctx->Tensor4ArgNameAndIndex("normalized_diff", 0);
    user_op::Tensor* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const bool has_beta_diff = beta_diff != nullptr;
    const bool has_gamma_diff = gamma_diff != nullptr;
    const bool has_normalized_diff = normalized_diff != nullptr;
    const bool has_gamma = gamma != nullptr;
    const int64_t begin_params_axis = ctx->Attr<int64_t>("begin_params_axis");
    const int64_t elem_cnt = dy->shape().elem_cnt();
    const int64_t m = dy->shape().Count(begin_params_axis);
    int max_active_blocks = 0;
    OF_CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks, LayerNormParamGradHalfImpl<int64_t>, GetLayerNormParamGradBlockSize(),
        GetParamGradDynamicSharedMemorySize<float16>(m)));
    if (has_gamma_diff && has_beta_diff && has_normalized_diff && max_active_blocks > 0) {
      const user_op::Tensor* normalized = ctx->Tensor4ArgNameAndIndex("normalized", 0);
      user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
      const int64_t num_blocks = GetLayerNormParamGradNumBlocks(dy->shape().elem_cnt());
      const size_t tmp_diff_size = GetCudaAlignedSize(num_blocks * m * sizeof(float16));
      float16* tmp_gamma_diff = tmp_buffer->mut_dptr<float16>();
      float16* tmp_beta_diff =
          reinterpret_cast<float16*>(tmp_buffer->mut_dptr<char>() + tmp_diff_size);
      float16* tmp_reduce_buf =
          reinterpret_cast<float16*>(tmp_buffer->mut_dptr<char>() + 2 * tmp_diff_size);
      CHECK_GE(tmp_buffer->shape().elem_cnt(), 3 * tmp_diff_size);
      if (elem_cnt > static_cast<int64_t>(GetMaxVal<int32_t>() / 2)) {
        LayerNormParamGradHalfImpl<int64_t>
            <<<GetLayerNormParamGradNumBlocks(elem_cnt), GetLayerNormParamGradBlockSize(),
               GetParamGradDynamicSharedMemorySize<float16>(m),
               ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
                elem_cnt, m, dy->dptr<half>(), normalized->dptr<half>(), gamma->dptr<half>(),
                reinterpret_cast<half*>(tmp_gamma_diff), reinterpret_cast<half*>(tmp_beta_diff),
                normalized_diff->mut_dptr<half>());
      } else {
        LayerNormParamGradHalfImpl<int32_t>
            <<<GetLayerNormParamGradNumBlocks(elem_cnt), GetLayerNormParamGradBlockSize(),
               GetParamGradDynamicSharedMemorySize<float16>(m),
               ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
                static_cast<int32_t>(elem_cnt), static_cast<int32_t>(m), dy->dptr<half>(),
                normalized->dptr<half>(), gamma->dptr<half>(),
                reinterpret_cast<half*>(tmp_gamma_diff), reinterpret_cast<half*>(tmp_beta_diff),
                normalized_diff->mut_dptr<half>());
      }
      NdUtil::ReduceSum(ctx->stream(), Var({1, m}, gamma_diff->mut_dptr<float16>()),
                        Val({num_blocks, m}, tmp_gamma_diff), Var({num_blocks, m}, tmp_reduce_buf));
      NdUtil::ReduceSum(ctx->stream(), Var({1, m}, beta_diff->mut_dptr<float16>()),
                        Val({num_blocks, m}, tmp_beta_diff), Var({num_blocks, m}, tmp_reduce_buf));
    } else {
      if (has_beta_diff) {
        user_op::Tensor* reduce_buf = ctx->Tensor4ArgNameAndIndex("reduce_buf", 0);
        CHECK_EQ(m, beta_diff->shape().elem_cnt());
        CHECK_EQ(dy->shape().elem_cnt() % m, 0);
        const int64_t n = dy->shape().elem_cnt() / m;
        NdUtil::ReduceSum(ctx->stream(), Var({1, m}, beta_diff->mut_dptr<float16>()),
                          Val({n, m}, dy->dptr<float16>()),
                          Var({n, m}, reduce_buf->mut_dptr<float16>()));
      }
      if (has_gamma_diff) {
        const user_op::Tensor* normalized = ctx->Tensor4ArgNameAndIndex("normalized", 0);
        user_op::Tensor* reduce_buf = ctx->Tensor4ArgNameAndIndex("reduce_buf", 0);
        CHECK_EQ(m, gamma_diff->shape().elem_cnt());
        CHECK_EQ(dy->shape().elem_cnt() % m, 0);
        const int64_t n = dy->shape().elem_cnt() / m;
        NdUtil::BroadcastMul(ctx->stream(), Var({n, m}, reduce_buf->mut_dptr<float16>()),
                             Val({n, m}, normalized->dptr<float16>()),
                             Val({n, m}, dy->dptr<float16>()));
        NdUtil::ReduceSum(ctx->stream(), Var({1, m}, gamma_diff->mut_dptr<float16>()),
                          Val({n, m}, reduce_buf->dptr<float16>()),
                          Var({n, m}, reduce_buf->mut_dptr<float16>()));
      }
      if (has_normalized_diff) {
        if (has_gamma) {
          CHECK_EQ(m, gamma->shape().elem_cnt());
          CHECK_EQ(dy->shape().elem_cnt() % m, 0);
          const int64_t n = dy->shape().elem_cnt() / m;
          NdUtil::BroadcastMul(ctx->stream(), Var({n, m}, normalized_diff->mut_dptr<float16>()),
                               Val({n, m}, dy->dptr<float16>()),
                               Val({1, m}, gamma->dptr<float16>()));
        } else {
          Memcpy<DeviceType::kCUDA>(ctx->stream(), normalized_diff->mut_dptr<void>(),
                                    dy->dptr<void>(),
                                    dy->shape().elem_cnt() * GetSizeOfDataType(dy->data_type()));
        }
      }
    }
  }
};

REGISTER_USER_KERNEL("layer_norm_param_grad")
    .SetCreateFn<LayerNormParamGradGpuHalfKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)
                     && (user_op::HobDataType("dy", 0) == DataType::kFloat16))
    .SetInferTmpSizeFn([](user_op::InferContext* ctx) {
      const int64_t begin_params_axis = ctx->Attr<int64_t>("begin_params_axis");
      const bool has_gamma_diff = ctx->has_output("gamma_diff", 0);
      const bool has_beta_diff = ctx->has_output("beta_diff", 0);
      const bool has_normalized_diff = ctx->has_output("normalized_diff", 0);
      const auto& dy = ctx->InputTensorDesc("dy", 0);
      const int64_t instance_size = dy.shape().Count(begin_params_axis);
      size_t tmp_buffer_size = 0;
      if (has_gamma_diff && has_beta_diff && has_normalized_diff) {
        const size_t tmp_gamma_diff =
            GetCudaAlignedSize(GetLayerNormParamGradNumBlocks(dy.shape().elem_cnt()) * instance_size
                               * sizeof(float16));
        const size_t tmp_beta_diff = tmp_gamma_diff;
        const size_t tmp_reduce_buf = tmp_gamma_diff;
        tmp_buffer_size = tmp_gamma_diff + tmp_beta_diff + tmp_reduce_buf;
      } else {
        tmp_buffer_size = 0;
      }
      return tmp_buffer_size;
    });
}  // namespace oneflow
