#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ndarray/ndarray_util.h"
#include "oneflow/core/cuda/atomic.cuh"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/ep/include/primitive/fill.h"
#include "oneflow/core/ep/include/primitive/matmul.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/layer_norm.cuh"
#if CUDA_VERSION >= 11000
#include <hip/hip_bf16.h>
#endif  // CUDA_VERSION >= 11000

namespace oneflow {

namespace {

template<typename SRC, typename DST, bool do_scale, bool do_center>
struct AffineStore {
  AffineStore(DST* y, int64_t row_size, const DST* gamma, const DST* beta)
      : y(y), row_size(row_size), gamma(gamma), beta(beta) {}
  template<int N>
  __device__ void store(const SRC* src, int64_t row, int64_t col) {
    cuda::layer_norm::Pack<DST, N> y_pack;
    cuda::layer_norm::Pack<DST, N> gamma_pack;
    cuda::layer_norm::Pack<DST, N> beta_pack;
    const int64_t offset = (row * row_size + col) / N;
    const int64_t gamma_offset = col / N;
    if (do_scale) {
      gamma_pack.storage =
          *(reinterpret_cast<const cuda::layer_norm::PackType<DST, N>*>(gamma) + gamma_offset);
    } else {
#pragma unroll
      for (int i = 0; i < N; ++i) { gamma_pack.elem[i] = static_cast<DST>(1.f); }
    }
    if (do_center) {
      beta_pack.storage =
          *(reinterpret_cast<const cuda::layer_norm::PackType<DST, N>*>(beta) + gamma_offset);
    } else {
#pragma unroll
      for (int i = 0; i < N; ++i) { beta_pack.elem[i] = static_cast<DST>(0.f); }
    }
#pragma unroll
    for (int i = 0; i < N; ++i) {
      DST normalized_i = static_cast<DST>(src[i]);
      if (do_scale || do_center) {
        y_pack.elem[i] = normalized_i * gamma_pack.elem[i] + beta_pack.elem[i];
      } else {
        y_pack.elem[i] = normalized_i;
      }
    }
    *(reinterpret_cast<cuda::layer_norm::PackType<DST, N>*>(y) + offset) = y_pack.storage;
  }
  DST* y;
  int64_t row_size;
  const DST* gamma;
  const DST* beta;
};

template<typename SRC, typename DST, bool do_scale>
struct ScaleLoad {
  using LoadType = DST;
  ScaleLoad(const SRC* src, const SRC* gamma, int64_t row_size)
      : src(src), gamma(gamma), row_size(row_size) {}
  template<int N>
  __device__ void load(DST* dst, int64_t row, int64_t col) const {
    cuda::layer_norm::Pack<SRC, N> src_pack;
    cuda::layer_norm::Pack<SRC, N> gamma_pack;
    const int64_t offset = (row * row_size + col) / N;
    const int64_t gamma_offset = col / N;
    src_pack.storage = *(reinterpret_cast<const cuda::layer_norm::PackType<SRC, N>*>(src) + offset);
    if (do_scale) {
      gamma_pack.storage =
          *(reinterpret_cast<const cuda::layer_norm::PackType<SRC, N>*>(gamma) + gamma_offset);
    } else {
#pragma unroll
      for (int i = 0; i < N; ++i) { gamma_pack.elem[i] = static_cast<SRC>(1.f); }
    }
#pragma unroll
    for (int i = 0; i < N; ++i) {
      dst[i] = static_cast<DST>(src_pack.elem[i] * gamma_pack.elem[i]);
    }
  }
  const SRC* src;
  const SRC* gamma;
  int64_t row_size;
};

template<typename SRC, typename DST, bool do_add>
struct AddStore {
  AddStore(const DST* add_to_output, DST* dst, int64_t row_size)
      : add_to_output(add_to_output), dst(dst), row_size(row_size) {}
  template<int N>
  __device__ void store(const SRC* src, int64_t row, int64_t col) {
    cuda::layer_norm::Pack<DST, N> add_to_output_pack;
    cuda::layer_norm::Pack<DST, N> dst_pack;
    const int64_t offset = (row * row_size + col) / N;
    if (do_add) {
      add_to_output_pack.storage =
          *(reinterpret_cast<const cuda::layer_norm::PackType<DST, N>*>(add_to_output) + offset);
    }
#pragma unroll
    for (int i = 0; i < N; ++i) {
      if (do_add) {
        dst_pack.elem[i] = static_cast<DST>(src[i]) + add_to_output_pack.elem[i];
      } else {
        dst_pack.elem[i] = static_cast<DST>(src[i]);
      }
    }
    *(reinterpret_cast<cuda::layer_norm::PackType<DST, N>*>(dst) + offset) = dst_pack.storage;
  }
  const DST* add_to_output;
  DST* dst;
  int64_t row_size;
};

template<typename T>
__inline__ __device__ T WarpReduce(T val) {
  for (int mask = 16; mask > 0; mask /= 2) { val += __shfl_down_sync(0xffffffff, val, mask); }
  return val;
}

constexpr int tile_size = 32;
constexpr int num_per_block = 4;
constexpr int block_dim_x = 32;
constexpr int block_dim_y = 32 / num_per_block;

template<typename T, typename ComputeType>
__global__ void LayerNormParamGrad(int rows, int cols, const T* __restrict__ dy,
                                   const T* __restrict__ x, const ComputeType* __restrict__ mean,
                                   const ComputeType* __restrict__ inv_var,
                                   T* __restrict__ tmp_gamma_diff, T* __restrict__ tmp_beta_diff) {
  __shared__ ComputeType dgamma[32][33];
  __shared__ ComputeType dbeta[32][33];
  ComputeType dgamma_sum[num_per_block];
  ComputeType dbeta_sum[num_per_block];
#pragma unroll
  for (int index = 0; index < num_per_block; ++index) {
    dgamma_sum[index] = 0;
    dbeta_sum[index] = 0;
  }
  const int col_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (col_id < cols) {
    for (int i = blockIdx.y * tile_size + threadIdx.y; i < rows; i += tile_size * gridDim.y) {
#pragma unroll
      for (int index = 0; index < num_per_block; ++index) {
        int row_id = i + index * blockDim.y;
        if (row_id < rows) {
          int offset = row_id * cols + col_id;
          const ComputeType dy_val = static_cast<ComputeType>(dy[offset]);
          const ComputeType x_val = static_cast<ComputeType>(x[offset]);
          const ComputeType mean_val = mean[row_id];
          const ComputeType inv_var_val = inv_var[row_id];
          dgamma_sum[index] += dy_val * (x_val - mean_val) * inv_var_val;
          dbeta_sum[index] += dy_val;
        }
      }
    }
  }
#pragma unroll
  for (int index = 0; index < num_per_block; ++index) {
    dgamma[index * blockDim.y + threadIdx.y][threadIdx.x] = dgamma_sum[index];
    dbeta[index * blockDim.y + threadIdx.y][threadIdx.x] = dbeta_sum[index];
  }
  __syncthreads();
#pragma unroll
  for (int index = 0; index < num_per_block; ++index) {
    const int col_id = blockIdx.x * blockDim.x + threadIdx.y + index * blockDim.y;
    if (col_id < cols) {
      ComputeType gamma_sum = dgamma[threadIdx.x][threadIdx.y + index * blockDim.y];
      ComputeType beta_sum = dbeta[threadIdx.x][threadIdx.y + index * blockDim.y];
      ComputeType global_dgamma = WarpReduce<ComputeType>(gamma_sum);
      ComputeType global_dbeta = WarpReduce<ComputeType>(beta_sum);
      if (threadIdx.x == 0) {
        const int offset = blockIdx.y * cols + col_id;
        tmp_gamma_diff[offset] = global_dgamma;
        tmp_beta_diff[offset] = global_dbeta;
      }
    }
  }
}

template<typename T>
int GetGirdDimY(const int64_t num_instances, const int64_t norm_size) {
  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  const int grid_dim_x = (norm_size + tile_size - 1) / tile_size;
  const int max_grid_dim_y = (num_instances + tile_size - 1) / tile_size;
  const int block_size = block_dim_x * block_dim_y;
  int max_active_blocks = 0;
  OF_CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &max_active_blocks, LayerNormParamGrad<T, ComputeType>, block_size, 0));
  int waves = 1;
  int dev;
  OF_CUDA_CHECK(hipGetDevice(&dev));
  int sm_count;
  OF_CUDA_CHECK(hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev));
  int num_blocks = max_active_blocks * sm_count * waves;
  int grid_dim_y = std::min(max_grid_dim_y, static_cast<int>(num_blocks / grid_dim_x));
  return std::max(grid_dim_y, 1);
}

template<typename T, bool do_scale, bool do_center>
void LayerNormForwardGpu(ep::Stream* stream, const int64_t num_instances, const int64_t norm_size,
                         const double epsilon, const T* x_ptr, const T* gamma_ptr,
                         const T* beta_ptr, T* y_ptr, user_op::Tensor* mean,
                         user_op::Tensor* inv_variance) {
  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  cuda::layer_norm::DirectLoad<T, T> load(x_ptr, norm_size);
  AffineStore<ComputeType, T, do_scale, do_center> store(y_ptr, norm_size, gamma_ptr, beta_ptr);
  cuda::layer_norm::DispatchLayerNorm<decltype(load), decltype(store), ComputeType>(
      stream->As<ep::CudaStream>()->hip_stream(), load, store, num_instances, norm_size, epsilon,
      mean->mut_dptr<ComputeType>(), inv_variance->mut_dptr<ComputeType>());
}

template<typename T>
void DispatchLayerNormForwardGpu(ep::Stream* stream, const int64_t num_instances,
                                 const int64_t norm_size, const double epsilon, const T* x_ptr,
                                 const T* gamma_ptr, const T* beta_ptr, T* y_ptr,
                                 user_op::Tensor* mean, user_op::Tensor* inv_variance) {
  if (gamma_ptr != nullptr && beta_ptr != nullptr) {
    LayerNormForwardGpu<T, true, true>(stream, num_instances, norm_size, epsilon, x_ptr, gamma_ptr,
                                       beta_ptr, y_ptr, mean, inv_variance);
  } else if (gamma_ptr != nullptr && beta_ptr == nullptr) {
    LayerNormForwardGpu<T, true, false>(stream, num_instances, norm_size, epsilon, x_ptr, gamma_ptr,
                                        beta_ptr, y_ptr, mean, inv_variance);
  } else if (gamma_ptr == nullptr && beta_ptr != nullptr) {
    LayerNormForwardGpu<T, false, true>(stream, num_instances, norm_size, epsilon, x_ptr, gamma_ptr,
                                        beta_ptr, y_ptr, mean, inv_variance);
  } else {
    LayerNormForwardGpu<T, false, false>(stream, num_instances, norm_size, epsilon, x_ptr,
                                         gamma_ptr, beta_ptr, y_ptr, mean, inv_variance);
  }
}

template<typename T, bool do_scale, bool do_add>
void LayerNormBackwardGpu(ep::Stream* stream, const int64_t num_instances, const int64_t norm_size,
                          const T* dy_ptr, const T* x_ptr, const user_op::Tensor* mean,
                          const user_op::Tensor* inv_variance, const T* gamma_ptr,
                          const T* add_to_output_ptr, T* dx_ptr) {
  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  cuda::layer_norm::DirectLoad<T, T> load_x(x_ptr, norm_size);
  ScaleLoad<T, T, do_scale> load_scaled_dy(dy_ptr, gamma_ptr, norm_size);
  AddStore<ComputeType, T, do_add> store(add_to_output_ptr, dx_ptr, norm_size);
  OF_CUDA_CHECK((cuda::layer_norm::DispatchLayerNormGrad<decltype(load_x), decltype(load_scaled_dy),
                                                         decltype(store), ComputeType>(
      stream->As<ep::CudaStream>()->hip_stream(), load_x, load_scaled_dy, store,
      mean->dptr<ComputeType>(), inv_variance->dptr<ComputeType>(), num_instances, norm_size)));
}

template<typename T, bool do_scale>
void DispatchLayerNormBackwardDoAdd(ep::Stream* stream, const int64_t num_instances,
                                    const int64_t norm_size, const T* dy_ptr, const T* x_ptr,
                                    const user_op::Tensor* mean,
                                    const user_op::Tensor* inv_variance, const T* gamma_ptr,
                                    const T* add_to_output_ptr, T* dx_ptr) {
  if (add_to_output_ptr != nullptr) {
    LayerNormBackwardGpu<T, do_scale, true>(stream, num_instances, norm_size, dy_ptr, x_ptr, mean,
                                            inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
  } else {
    LayerNormBackwardGpu<T, do_scale, false>(stream, num_instances, norm_size, dy_ptr, x_ptr, mean,
                                             inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
  }
}

template<typename T>
void LaunchLayerNormBackward(ep::Stream* stream, const int64_t num_instances,
                             const int64_t norm_size, const T* dy_ptr, const T* x_ptr,
                             const user_op::Tensor* mean, const user_op::Tensor* inv_variance,
                             const T* gamma_ptr, const T* add_to_output_ptr, T* dx_ptr) {
  if (gamma_ptr != nullptr) {
    DispatchLayerNormBackwardDoAdd<T, true>(stream, num_instances, norm_size, dy_ptr, x_ptr, mean,
                                            inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
  } else {
    DispatchLayerNormBackwardDoAdd<T, false>(stream, num_instances, norm_size, dy_ptr, x_ptr, mean,
                                             inv_variance, gamma_ptr, add_to_output_ptr, dx_ptr);
  }
}

}  // namespace

template<typename T>
class LayerNormGpuKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  LayerNormGpuKernel() = default;
  ~LayerNormGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    const double epsilon = ctx->Attr<double>("epsilon");
    CHECK_GE(epsilon, HIPDNN_BN_MIN_EPSILON);
    const int64_t num_instances = mean->shape_view().elem_cnt();
    const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
    const T* gamma_ptr = nullptr;
    const T* beta_ptr = nullptr;
    if (ctx->has_input("gamma", 0)) {
      const user_op::Tensor* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
      gamma_ptr = gamma->dptr<T>();
      CHECK_EQ(gamma->shape_view().elem_cnt(), norm_size);
    }
    if (ctx->has_input("beta", 0)) { beta_ptr = ctx->Tensor4ArgNameAndIndex("beta", 0)->dptr<T>(); }
    DispatchLayerNormForwardGpu<T>(ctx->stream(), num_instances, norm_size, epsilon, x->dptr<T>(),
                                   gamma_ptr, beta_ptr, y->mut_dptr<T>(), mean, inv_variance);
  };
};

#define REGISTER_LAYER_NORM_CUDA_KERNEL(dtype)                         \
  REGISTER_USER_KERNEL("layer_norm")                                   \
      .SetCreateFn<LayerNormGpuKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value));

REGISTER_LAYER_NORM_CUDA_KERNEL(float)
REGISTER_LAYER_NORM_CUDA_KERNEL(double)
REGISTER_LAYER_NORM_CUDA_KERNEL(half)
#if CUDA_VERSION >= 11000
REGISTER_LAYER_NORM_CUDA_KERNEL(hip_bfloat16)
#endif

template<typename T>
class LayerNormGradGpuKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  LayerNormGradGpuKernel() = default;
  ~LayerNormGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const int64_t num_instances = mean->shape_view().elem_cnt();
    const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
    const T* gamma_ptr = nullptr;
    if (ctx->has_input("gamma", 0)) {
      gamma_ptr = ctx->Tensor4ArgNameAndIndex("gamma", 0)->dptr<T>();
    }
    const T* add_to_output_ptr = nullptr;
    if (ctx->has_input("_add_to_output", 0)) {
      const user_op::Tensor* add_to_output = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      CHECK_EQ(add_to_output->data_type(), dx->data_type());
      CHECK_EQ(add_to_output->shape_view(), dx->shape_view());
      add_to_output_ptr = add_to_output->dptr<T>();
    }
    LaunchLayerNormBackward<T>(ctx->stream(), num_instances, norm_size, dy->dptr<T>(), x->dptr<T>(),
                               mean, inv_variance, gamma_ptr, add_to_output_ptr, dx->mut_dptr<T>());
  };
};

#define REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(dtype)                                        \
  REGISTER_USER_KERNEL("layer_norm_grad")                                                  \
      .SetCreateFn<LayerNormGradGpuKernel<dtype>>()                                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                     \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value))    \
      .SetInplaceProposalFn(                                                               \
          [](const user_op::InferContext& ctx,                                             \
             const user_op::AddInplaceArgPair& AddInplaceArgPairFn) -> Maybe<void> {       \
            if (ctx.has_input("_add_to_output", 0)) {                                      \
              OF_RETURN_IF_ERROR(AddInplaceArgPairFn("dx", 0, "_add_to_output", 0, true)); \
            }                                                                              \
            return Maybe<void>::Ok();                                                      \
          });

REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(float)
REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(double)
REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(half)
#if CUDA_VERSION >= 11000
REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(hip_bfloat16)
#endif

template<typename T>
class LayerNormParamGradGpuKernel final : public user_op::OpKernel,
                                          public user_op::CudaGraphSupport {
 public:
  LayerNormParamGradGpuKernel() = default;
  ~LayerNormParamGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    const int64_t num_instances = mean->shape_view().elem_cnt();
    const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const DataType data_type = dy->data_type();
    const int grid_dim_x = (norm_size + tile_size - 1) / tile_size;
    const int grid_dim_y = GetGirdDimY<T>(num_instances, norm_size);
    const size_t tmp_gamma_diff_size = grid_dim_y * norm_size * sizeof(T);
    T* tmp_gamma_diff_ptr = reinterpret_cast<T*>(tmp_buffer->mut_dptr());
    T* tmp_beta_diff_ptr = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + tmp_gamma_diff_size);
    T* reduce_buf_ptr =
        reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + 2 * tmp_gamma_diff_size);
    using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
    LayerNormParamGrad<T, ComputeType><<<dim3(grid_dim_x, grid_dim_y), dim3(32, 32 / num_per_block),
                                         0, ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        num_instances, norm_size, dy->dptr<T>(), x->dptr<T>(), mean->dptr<ComputeType>(),
        inv_variance->dptr<ComputeType>(), tmp_gamma_diff_ptr, tmp_beta_diff_ptr);
    const int32_t m = norm_size;
    const int32_t n = 1;
    const int32_t k = grid_dim_y;
    std::unique_ptr<ep::primitive::Fill> fill =
        ep::primitive::NewPrimitive<ep::primitive::FillFactory>(ctx->stream()->device_type(),
                                                                data_type);
    CHECK(fill);
    fill->Launch(ctx->stream(), reduce_buf_ptr, 1.0, grid_dim_y);
    std::unique_ptr<ep::primitive::Matmul> matmul =
        ep::primitive::NewPrimitive<ep::primitive::MatmulFactory>(
            ctx->stream()->device_type(), data_type, ep::primitive::BlasTransposeType::T,
            ep::primitive::BlasTransposeType::N);
    CHECK(matmul);
    if (ctx->has_output("gamma_diff", 0)) {
      user_op::Tensor* gamma_diff = ctx->Tensor4ArgNameAndIndex("gamma_diff", 0);
      matmul->Launch(ctx->stream(), m, n, k, 1.0, tmp_gamma_diff_ptr, reduce_buf_ptr, 0.0,
                     gamma_diff->mut_dptr());
    }
    if (ctx->has_output("beta_diff", 0)) {
      user_op::Tensor* beta_diff = ctx->Tensor4ArgNameAndIndex("beta_diff", 0);
      matmul->Launch(ctx->stream(), m, n, k, 1.0, tmp_beta_diff_ptr, reduce_buf_ptr, 0.0,
                     beta_diff->mut_dptr());
    }
  };
};

#define REGISTER_LAYER_NORM_PARAM_GRAD_GPU_KERNEL(dtype)                                    \
  REGISTER_USER_KERNEL("layer_norm_param_grad")                                             \
      .SetCreateFn<LayerNormParamGradGpuKernel<dtype>>()                                    \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                      \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value))     \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                   \
        const int64_t begin_params_axis = ctx->Attr<int64_t>("begin_params_axis");          \
        const bool has_gamma_diff = ctx->has_output("gamma_diff", 0);                       \
        const bool has_beta_diff = ctx->has_output("beta_diff", 0);                         \
        const auto& dy = ctx->InputTensorDesc("dy", 0);                                     \
        const int64_t num_instances = dy.shape().Count(0, begin_params_axis);               \
        const int64_t norm_size = dy.shape().Count(begin_params_axis);                      \
        const int grid_dim_y = GetGirdDimY<dtype>(num_instances, norm_size);                \
        size_t tmp_buffer_size = (2 * grid_dim_y * norm_size + grid_dim_y) * sizeof(dtype); \
        return tmp_buffer_size;                                                             \
      });

REGISTER_LAYER_NORM_PARAM_GRAD_GPU_KERNEL(float)
REGISTER_LAYER_NORM_PARAM_GRAD_GPU_KERNEL(double)
REGISTER_LAYER_NORM_PARAM_GRAD_GPU_KERNEL(half)
#if CUDA_VERSION >= 11000
REGISTER_LAYER_NORM_PARAM_GRAD_GPU_KERNEL(hip_bfloat16)
#endif

}  // namespace oneflow
