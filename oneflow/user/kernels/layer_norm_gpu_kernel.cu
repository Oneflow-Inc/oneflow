#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ndarray/ndarray_util.h"
#include "oneflow/core/cuda/atomic.cuh"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/ep/include/primitive/fill.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/layer_norm.cuh"

namespace oneflow {

namespace {

std::unique_ptr<ep::primitive::Fill> NewFillPrimitive(ep::Stream* stream, DataType data_type) {
  std::unique_ptr<ep::primitive::Fill> fill =
      ep::primitive::NewPrimitive<ep::primitive::FillFactory>(stream->device_type(), data_type);
  CHECK(fill);
  return fill;
}

class LayerNormCudnnBnCtx final {
 public:
  LayerNormCudnnBnCtx(const ShapeView& data_shape, const ShapeView& param_shape,
                      DataType data_type) {
    const int64_t cudnn_c = param_shape.elem_cnt();
    CHECK_EQ(data_shape.elem_cnt() % cudnn_c, 0);
    const int64_t cudnn_w = data_shape.elem_cnt() / cudnn_c;
    CHECK_LT(cudnn_c, GetMaxVal<int32_t>());
    CHECK_LT(cudnn_w, GetMaxVal<int32_t>());
    data_tensor_desc_.reset(new CudnnTensorDesc(HIPDNN_TENSOR_NCHW, data_type, 1,
                                                static_cast<int32_t>(cudnn_c), 1,
                                                static_cast<int32_t>(cudnn_w)));
    DataType param_dtype = data_type == DataType::kFloat16 ? DataType::kFloat : data_type;
    param_tensor_desc_.reset(new CudnnTensorDesc(HIPDNN_TENSOR_NCHW, param_dtype, 1,
                                                 static_cast<int32_t>(cudnn_c), 1, 1));
#if (CUDNN_VERSION >= 7000)
    mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;
#else
    mode_ = HIPDNN_BATCHNORM_SPATIAL;
#endif
  }
  ~LayerNormCudnnBnCtx() = default;

  const hipdnnTensorDescriptor_t& data_tensor_desc() const { return data_tensor_desc_->Get(); }
  const hipdnnTensorDescriptor_t& param_tensor_desc() const { return param_tensor_desc_->Get(); }
  hipdnnBatchNormMode_t mode() const { return mode_; };

 private:
  std::unique_ptr<CudnnTensorDesc> data_tensor_desc_;
  std::unique_ptr<CudnnTensorDesc> param_tensor_desc_;
  hipdnnBatchNormMode_t mode_;
};

template<typename SRC, typename DST, bool do_scale, bool do_center>
struct AffineStore {
  AffineStore(DST* normalized, DST* y, int64_t row_size, const DST* gamma, const DST* beta)
      : normalized(normalized), y(y), row_size(row_size), gamma(gamma), beta(beta) {}
  template<int N>
  __device__ void store(const SRC* src, int64_t row, int64_t col) {
    cuda::layer_norm::Pack<DST, N> y_pack;
    cuda::layer_norm::Pack<DST, N> normalized_pack;
    cuda::layer_norm::Pack<DST, N> gamma_pack;
    cuda::layer_norm::Pack<DST, N> beta_pack;
    const int64_t offset = row * row_size + col;
    if (do_scale) {
      gamma_pack.storage =
          *reinterpret_cast<const cuda::layer_norm::PackType<DST, N>*>(gamma + col);
    } else {
#pragma unroll
      for (int i = 0; i < N; ++i) { gamma_pack.elem[i] = 1; }
    }
    if (do_center) {
      beta_pack.storage = *reinterpret_cast<const cuda::layer_norm::PackType<DST, N>*>(beta + col);
    } else {
#pragma unroll
      for (int i = 0; i < N; ++i) { beta_pack.elem[i] = 0; }
    }
#pragma unroll
    for (int i = 0; i < N; ++i) {
      DST normalized_i = static_cast<DST>(src[i]);
      if (do_scale) { normalized_pack.elem[i] = normalized_i; }
      if (do_scale || do_center) {
        y_pack.elem[i] = normalized_i * gamma_pack.elem[i] + beta_pack.elem[i];
      } else {
        y_pack.elem[i] = normalized_i;
      }
    }
    *reinterpret_cast<cuda::layer_norm::PackType<DST, N>*>(y + offset) = y_pack.storage;
    if (do_scale) {
      *reinterpret_cast<cuda::layer_norm::PackType<DST, N>*>(normalized + offset) =
          normalized_pack.storage;
    }
  }
  DST* normalized;
  DST* y;
  int64_t row_size;
  const DST* gamma;
  const DST* beta;
};

constexpr int64_t kLayerNormParamGradGpuBlockSize = 512;

int64_t GetLayerNormParamGradBlockSize() { return kLayerNormParamGradGpuBlockSize; }

int64_t GetLayerNormParamGradNumBlocks(const int64_t elem_cnt) {
  return std::min(static_cast<int>((elem_cnt + kLayerNormParamGradGpuBlockSize - 1)
                                   / kLayerNormParamGradGpuBlockSize),
                  256);
}

template<typename T>
int64_t GetParamGradDynamicSharedMemorySize(const int64_t instance_size) {
  return 2 * instance_size * sizeof(T);
}

template<>
int64_t GetParamGradDynamicSharedMemorySize<float16>(const int64_t instance_size) {
  return 2 * instance_size * sizeof(float);
}

template<typename T, typename I>
__global__ void LayerNormParamGradImpl(const I n, const I instance_size, const T* dy,
                                       const T* normalized, const T* gamma, T* gamma_diff,
                                       T* beta_diff, T* normalized_diff) {
  extern __shared__ __align__(sizeof(double)) unsigned char bw_shared_buf[];
  auto* gamma_diff_sum_buf = reinterpret_cast<T*>(bw_shared_buf);
  auto* beta_diff_sum_buf = gamma_diff_sum_buf + instance_size;
  const I tid = threadIdx.x;
  for (I elem_id = tid; elem_id < instance_size; elem_id += blockDim.x) {
    gamma_diff_sum_buf[elem_id] = 0;
    beta_diff_sum_buf[elem_id] = 0;
  }
  __syncthreads();
  CUDA_1D_KERNEL_LOOP_T(I, i, n) {
    const I elem_id = i % instance_size;
    T dy_val = dy[i];
    T normalized_val = normalized[i];
    cuda::atomic::Add(&gamma_diff_sum_buf[elem_id], dy_val * normalized_val);
    cuda::atomic::Add(&beta_diff_sum_buf[elem_id], dy_val);
    T gamma_val = gamma[elem_id];
    normalized_diff[i] = gamma_val * dy_val;
  }
  __syncthreads();
  for (I elem_id = tid; elem_id < instance_size; elem_id += blockDim.x) {
    cuda::atomic::Add(gamma_diff + elem_id, gamma_diff_sum_buf[elem_id]);
    cuda::atomic::Add(beta_diff + elem_id, beta_diff_sum_buf[elem_id]);
  }
}

template<typename I>
__global__ void LayerNormParamGradHalfImpl(const I n, const I instance_size, const half* dy,
                                           const half* normalized, const half* gamma,
                                           half* tmp_gamma_diff, half* tmp_beta_diff,
                                           half* normalized_diff) {
  extern __shared__ __align__(sizeof(double)) unsigned char bw_shared_buf[];
  auto* gamma_diff_sum_buf = reinterpret_cast<float*>(bw_shared_buf);
  auto* beta_diff_sum_buf = gamma_diff_sum_buf + instance_size;
  const I tid = threadIdx.x;
  for (I elem_id = tid; elem_id < instance_size; elem_id += blockDim.x) {
    gamma_diff_sum_buf[elem_id] = 0;
    beta_diff_sum_buf[elem_id] = 0;
  }
  __syncthreads();
  CUDA_1D_KERNEL_LOOP_T(I, i, n) {
    const I elem_id = i % instance_size;
    half dy_val = dy[i];
    half normalized_val = normalized[i];
    cuda::atomic::Add(&gamma_diff_sum_buf[elem_id],
                      __half2float(dy_val) * __half2float(normalized_val));
    cuda::atomic::Add(&beta_diff_sum_buf[elem_id], __half2float(dy_val));
    half gamma_val = gamma[elem_id];
    normalized_diff[i] = __hmul(gamma_val, dy_val);
  }
  __syncthreads();
  for (I elem_id = tid; elem_id < instance_size; elem_id += blockDim.x) {
    const I offset = blockIdx.x * instance_size + elem_id;
    tmp_gamma_diff[offset] = __float2half(gamma_diff_sum_buf[elem_id]);
    tmp_beta_diff[offset] = __float2half(beta_diff_sum_buf[elem_id]);
  }
}

template<typename T, bool do_scale, bool do_center>
void LayerNormForwardGpu(ep::Stream* stream, const int64_t num_instances, const int64_t norm_size,
                         const double epsilon, const T* x_ptr, const T* gamma_ptr,
                         const T* beta_ptr, T* normalized_ptr, T* y_ptr, user_op::Tensor* mean,
                         user_op::Tensor* inv_variance) {
  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  cuda::layer_norm::DirectLoad<T, ComputeType> load(x_ptr, norm_size);
  AffineStore<ComputeType, T, do_scale, do_center> store(normalized_ptr, y_ptr, norm_size,
                                                         gamma_ptr, beta_ptr);
  cuda::layer_norm::DispatchLayerNorm<decltype(load), decltype(store), ComputeType>(
      stream->As<ep::CudaStream>()->hip_stream(), load, store, num_instances, norm_size, epsilon,
      mean->mut_dptr<ComputeType>(), inv_variance->mut_dptr<ComputeType>());
}

template<typename T>
void DispatchLayerNormForwardGpu(ep::Stream* stream, const int64_t num_instances,
                                 const int64_t norm_size, const double epsilon, const T* x_ptr,
                                 const T* gamma_ptr, const T* beta_ptr, T* normalized_ptr, T* y_ptr,
                                 user_op::Tensor* mean, user_op::Tensor* inv_variance) {
  if (gamma_ptr != nullptr && beta_ptr != nullptr) {
    LayerNormForwardGpu<T, true, true>(stream, num_instances, norm_size, epsilon, x_ptr, gamma_ptr,
                                       beta_ptr, normalized_ptr, y_ptr, mean, inv_variance);
  } else if (gamma_ptr != nullptr && beta_ptr == nullptr) {
    LayerNormForwardGpu<T, true, false>(stream, num_instances, norm_size, epsilon, x_ptr, gamma_ptr,
                                        beta_ptr, normalized_ptr, y_ptr, mean, inv_variance);
  } else if (gamma_ptr == nullptr && beta_ptr != nullptr) {
    LayerNormForwardGpu<T, false, true>(stream, num_instances, norm_size, epsilon, x_ptr, gamma_ptr,
                                        beta_ptr, normalized_ptr, y_ptr, mean, inv_variance);
  } else {
    LayerNormForwardGpu<T, false, false>(stream, num_instances, norm_size, epsilon, x_ptr,
                                         gamma_ptr, beta_ptr, normalized_ptr, y_ptr, mean,
                                         inv_variance);
  }
}

}  // namespace

template<typename T>
class LayerNormGpuKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  LayerNormGpuKernel() = default;
  ~LayerNormGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    user_op::Tensor* normalized =
        ctx->has_input("gamma", 0) ? ctx->Tensor4ArgNameAndIndex("normalized", 0) : y;
    const double epsilon = ctx->Attr<double>("epsilon");
    CHECK_GE(epsilon, HIPDNN_BN_MIN_EPSILON);
    const int64_t num_instances = mean->shape().elem_cnt();
    const int64_t norm_size = x->shape().elem_cnt() / num_instances;
    const T* gamma_ptr = nullptr;
    const T* beta_ptr = nullptr;
    if (ctx->has_input("gamma", 0)) {
      const user_op::Tensor* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
      gamma_ptr = gamma->dptr<T>();
      CHECK_EQ(gamma->shape().elem_cnt(), norm_size);
    }
    if (ctx->has_input("beta", 0)) { beta_ptr = ctx->Tensor4ArgNameAndIndex("beta", 0)->dptr<T>(); }
    DispatchLayerNormForwardGpu<T>(ctx->stream(), num_instances, norm_size, epsilon, x->dptr<T>(),
                                   gamma_ptr, beta_ptr, normalized->mut_dptr<T>(), y->mut_dptr<T>(),
                                   mean, inv_variance);
  };
};

#define REGISTER_LAYER_NORM_CUDA_KERNEL(dtype)                         \
  REGISTER_USER_KERNEL("layer_norm")                                   \
      .SetCreateFn<LayerNormGpuKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value));

REGISTER_LAYER_NORM_CUDA_KERNEL(float)
REGISTER_LAYER_NORM_CUDA_KERNEL(double)
REGISTER_LAYER_NORM_CUDA_KERNEL(half)

template<typename T, typename BNParamT>
class LayerNormGradGpuKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  LayerNormGradGpuKernel() = default;
  ~LayerNormGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const size_t aligned_buffer_size =
        GetCudaAlignedSize(mean->shape().elem_cnt() * GetSizeOfDataType(mean->data_type()));
    char* cudnn_bn_scale_ones_dptr = tmp_buffer->mut_dptr<char>();
    char* cudnn_bn_scale_diff_buf_dptr = cudnn_bn_scale_ones_dptr + aligned_buffer_size;
    char* cudnn_bn_bias_diff_buf_dptr = cudnn_bn_scale_ones_dptr + aligned_buffer_size;
    auto fill = NewFillPrimitive(ctx->stream(), mean->data_type());
    fill->Launch(ctx->stream(), cudnn_bn_scale_ones_dptr, 1, mean->shape().elem_cnt());
    const void* sp_alpha = CudnnSPOnePtr<T>();
    const void* sp_beta = nullptr;
    if (ctx->has_input("_add_to_output", 0)) {
      const user_op::Tensor* add_to_output = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      CHECK_EQ(add_to_output->data_type(), dx->data_type());
      CHECK_EQ(add_to_output->shape(), dx->shape());
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), dx->mut_dptr<void>(), add_to_output->dptr<void>(),
          add_to_output->shape().elem_cnt() * GetSizeOfDataType(add_to_output->data_type()));
      sp_beta = CudnnSPOnePtr<T>();
    } else {
      sp_beta = CudnnSPZeroPtr<T>();
    }
    const double epsilon = ctx->Attr<double>("epsilon");
    CHECK_GE(epsilon, HIPDNN_BN_MIN_EPSILON);
    LayerNormCudnnBnCtx bn_ctx(x->shape(), mean->shape(), x->data_type());
    OF_CUDNN_CHECK(hipdnnBatchNormalizationBackward(
        ctx->stream()->As<ep::CudaStream>()->cudnn_handle(), bn_ctx.mode(), sp_alpha, sp_beta,
        CudnnSPOnePtr<T>(), CudnnSPZeroPtr<T>(), bn_ctx.data_tensor_desc(), x->dptr<T>(),
        bn_ctx.data_tensor_desc(), dy->dptr<T>(), bn_ctx.data_tensor_desc(), dx->mut_dptr<T>(),
        bn_ctx.param_tensor_desc(), reinterpret_cast<const BNParamT*>(cudnn_bn_scale_ones_dptr),
        reinterpret_cast<BNParamT*>(cudnn_bn_scale_diff_buf_dptr),
        reinterpret_cast<BNParamT*>(cudnn_bn_bias_diff_buf_dptr), epsilon, mean->dptr(),
        inv_variance->dptr()));
  };
};

#define REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(dtype, bn_param_dtype)                        \
  REGISTER_USER_KERNEL("layer_norm_grad")                                                  \
      .SetCreateFn<LayerNormGradGpuKernel<dtype, bn_param_dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                     \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value))    \
      .SetInferTmpSizeFn([](oneflow::user_op::InferContext* ctx) {                         \
        const user_op::TensorDesc& mean = ctx->InputTensorDesc("mean", 0);                 \
        const DataType& data_type = mean.data_type();                                      \
        const int64_t elem_cnt = mean.shape().elem_cnt();                                  \
        return GetCudaAlignedSize(elem_cnt * GetSizeOfDataType(data_type)) * 3;            \
      })                                                                                   \
      .SetInplaceProposalFn(                                                               \
          [](const user_op::InferContext& ctx,                                             \
             const user_op::AddInplaceArgPair& AddInplaceArgPairFn) -> Maybe<void> {       \
            if (ctx.has_input("_add_to_output", 0)) {                                      \
              OF_RETURN_IF_ERROR(AddInplaceArgPairFn("dx", 0, "_add_to_output", 0, true)); \
            }                                                                              \
            return Maybe<void>::Ok();                                                      \
          });

REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(float, float)
REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(double, double)
REGISTER_LAYER_NORM_GRAD_CUDA_KERNEL(float16, float)

template<typename T>
class LayerNormParamGradGpuKernel final : public user_op::OpKernel,
                                          public user_op::CudaGraphSupport {
 public:
  LayerNormParamGradGpuKernel() = default;
  ~LayerNormParamGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    using NdUtil = NdarrayUtil<DeviceType::kCUDA, T>;
    auto Val = NdUtil::GetValNdarrayBuilder();
    auto Var = NdUtil::GetVarNdarrayBuilder();
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* beta_diff = ctx->Tensor4ArgNameAndIndex("beta_diff", 0);
    user_op::Tensor* gamma_diff = ctx->Tensor4ArgNameAndIndex("gamma_diff", 0);
    user_op::Tensor* normalized_diff = ctx->Tensor4ArgNameAndIndex("normalized_diff", 0);
    user_op::Tensor* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const bool has_beta_diff = beta_diff != nullptr;
    const bool has_gamma_diff = gamma_diff != nullptr;
    const bool has_normalized_diff = normalized_diff != nullptr;
    const bool has_gamma = gamma != nullptr;
    const int64_t begin_params_axis = ctx->Attr<int64_t>("begin_params_axis");
    const int64_t elem_cnt = dy->shape().elem_cnt();
    const int64_t m = dy->shape().Count(begin_params_axis);
    int max_active_blocks = 0;
    OF_CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks, LayerNormParamGradImpl<T, int64_t>, GetLayerNormParamGradBlockSize(),
        GetParamGradDynamicSharedMemorySize<T>(m)));
    if (has_gamma_diff && has_beta_diff && has_normalized_diff && max_active_blocks > 0) {
      const user_op::Tensor* normalized = ctx->Tensor4ArgNameAndIndex("normalized", 0);
      Memset<DeviceType::kCUDA>(ctx->stream(), gamma_diff->mut_dptr<T>(), 0,
                                gamma_diff->shape().elem_cnt() * sizeof(T));
      Memset<DeviceType::kCUDA>(ctx->stream(), beta_diff->mut_dptr<T>(), 0,
                                beta_diff->shape().elem_cnt() * sizeof(T));
      if (elem_cnt > static_cast<int64_t>(GetMaxVal<int32_t>() / 2)) {
        LayerNormParamGradImpl<T, int64_t>
            <<<GetLayerNormParamGradNumBlocks(elem_cnt), GetLayerNormParamGradBlockSize(),
               GetParamGradDynamicSharedMemorySize<T>(m),
               ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
                elem_cnt, m, dy->dptr<T>(), normalized->dptr<T>(), gamma->dptr<T>(),
                gamma_diff->mut_dptr<T>(), beta_diff->mut_dptr<T>(),
                normalized_diff->mut_dptr<T>());
      } else {
        LayerNormParamGradImpl<T, int32_t>
            <<<GetLayerNormParamGradNumBlocks(elem_cnt), GetLayerNormParamGradBlockSize(),
               GetParamGradDynamicSharedMemorySize<T>(m),
               ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
                static_cast<int32_t>(elem_cnt), static_cast<int32_t>(m), dy->dptr<T>(),
                normalized->dptr<T>(), gamma->dptr<T>(), gamma_diff->mut_dptr<T>(),
                beta_diff->mut_dptr<T>(), normalized_diff->mut_dptr<T>());
      }
    } else {
      if (has_beta_diff) {
        user_op::Tensor* reduce_buf = ctx->Tensor4ArgNameAndIndex("reduce_buf", 0);
        CHECK_EQ(m, beta_diff->shape().elem_cnt());
        CHECK_EQ(dy->shape().elem_cnt() % m, 0);
        const int64_t n = dy->shape().elem_cnt() / m;
        NdUtil::ReduceSum(ctx->stream(), Var({1, m}, beta_diff->mut_dptr<T>()),
                          Val({n, m}, dy->dptr<T>()), Var({n, m}, reduce_buf->mut_dptr<T>()));
      }
      if (has_gamma_diff) {
        const user_op::Tensor* normalized = ctx->Tensor4ArgNameAndIndex("normalized", 0);
        user_op::Tensor* reduce_buf = ctx->Tensor4ArgNameAndIndex("reduce_buf", 0);
        CHECK_EQ(m, gamma_diff->shape().elem_cnt());
        CHECK_EQ(dy->shape().elem_cnt() % m, 0);
        const int64_t n = dy->shape().elem_cnt() / m;
        NdUtil::BroadcastMul(ctx->stream(), Var({n, m}, reduce_buf->mut_dptr<T>()),
                             Val({n, m}, normalized->dptr<T>()), Val({n, m}, dy->dptr<T>()));
        NdUtil::ReduceSum(ctx->stream(), Var({1, m}, gamma_diff->mut_dptr<T>()),
                          Val({n, m}, reduce_buf->dptr<T>()),
                          Var({n, m}, reduce_buf->mut_dptr<T>()));
      }
      if (has_normalized_diff) {
        if (has_gamma) {
          CHECK_EQ(m, gamma->shape().elem_cnt());
          CHECK_EQ(dy->shape().elem_cnt() % m, 0);
          const int64_t n = dy->shape().elem_cnt() / m;
          NdUtil::BroadcastMul(ctx->stream(), Var({n, m}, normalized_diff->mut_dptr<T>()),
                               Val({n, m}, dy->dptr<T>()), Val({1, m}, gamma->dptr<T>()));
        } else {
          Memcpy<DeviceType::kCUDA>(ctx->stream(), normalized_diff->mut_dptr<void>(),
                                    dy->dptr<void>(),
                                    dy->shape().elem_cnt() * GetSizeOfDataType(dy->data_type()));
        }
      }
    }
  };
};

#define REGISTER_LAYER_NORM_PARAM_GRAD_CUDA_KERNEL(dtype)              \
  REGISTER_USER_KERNEL("layer_norm_param_grad")                        \
      .SetCreateFn<LayerNormParamGradGpuKernel<dtype>>()               \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value));

REGISTER_LAYER_NORM_PARAM_GRAD_CUDA_KERNEL(float)
REGISTER_LAYER_NORM_PARAM_GRAD_CUDA_KERNEL(double)

class LayerNormParamGradGpuHalfKernel final : public user_op::OpKernel,
                                              public user_op::CudaGraphSupport {
 public:
  LayerNormParamGradGpuHalfKernel() = default;
  ~LayerNormParamGradGpuHalfKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    using NdUtil = NdarrayUtil<DeviceType::kCUDA, float16>;
    auto Val = NdUtil::GetValNdarrayBuilder();
    auto Var = NdUtil::GetVarNdarrayBuilder();
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* beta_diff = ctx->Tensor4ArgNameAndIndex("beta_diff", 0);
    user_op::Tensor* gamma_diff = ctx->Tensor4ArgNameAndIndex("gamma_diff", 0);
    user_op::Tensor* normalized_diff = ctx->Tensor4ArgNameAndIndex("normalized_diff", 0);
    user_op::Tensor* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
    const bool has_beta_diff = beta_diff != nullptr;
    const bool has_gamma_diff = gamma_diff != nullptr;
    const bool has_normalized_diff = normalized_diff != nullptr;
    const bool has_gamma = gamma != nullptr;
    const int64_t begin_params_axis = ctx->Attr<int64_t>("begin_params_axis");
    const int64_t elem_cnt = dy->shape().elem_cnt();
    const int64_t m = dy->shape().Count(begin_params_axis);
    int max_active_blocks = 0;
    OF_CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks, LayerNormParamGradHalfImpl<int64_t>, GetLayerNormParamGradBlockSize(),
        GetParamGradDynamicSharedMemorySize<float16>(m)));
    if (has_gamma_diff && has_beta_diff && has_normalized_diff && max_active_blocks > 0) {
      const user_op::Tensor* normalized = ctx->Tensor4ArgNameAndIndex("normalized", 0);
      user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
      const int64_t num_blocks = GetLayerNormParamGradNumBlocks(dy->shape().elem_cnt());
      const size_t tmp_diff_size = GetCudaAlignedSize(num_blocks * m * sizeof(float16));
      float16* tmp_gamma_diff = tmp_buffer->mut_dptr<float16>();
      float16* tmp_beta_diff =
          reinterpret_cast<float16*>(tmp_buffer->mut_dptr<char>() + tmp_diff_size);
      float16* tmp_reduce_buf =
          reinterpret_cast<float16*>(tmp_buffer->mut_dptr<char>() + 2 * tmp_diff_size);
      CHECK_GE(tmp_buffer->shape().elem_cnt(), 3 * tmp_diff_size);
      if (elem_cnt > static_cast<int64_t>(GetMaxVal<int32_t>() / 2)) {
        LayerNormParamGradHalfImpl<int64_t>
            <<<GetLayerNormParamGradNumBlocks(elem_cnt), GetLayerNormParamGradBlockSize(),
               GetParamGradDynamicSharedMemorySize<float16>(m),
               ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
                elem_cnt, m, dy->dptr<half>(), normalized->dptr<half>(), gamma->dptr<half>(),
                reinterpret_cast<half*>(tmp_gamma_diff), reinterpret_cast<half*>(tmp_beta_diff),
                normalized_diff->mut_dptr<half>());
      } else {
        LayerNormParamGradHalfImpl<int32_t>
            <<<GetLayerNormParamGradNumBlocks(elem_cnt), GetLayerNormParamGradBlockSize(),
               GetParamGradDynamicSharedMemorySize<float16>(m),
               ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
                static_cast<int32_t>(elem_cnt), static_cast<int32_t>(m), dy->dptr<half>(),
                normalized->dptr<half>(), gamma->dptr<half>(),
                reinterpret_cast<half*>(tmp_gamma_diff), reinterpret_cast<half*>(tmp_beta_diff),
                normalized_diff->mut_dptr<half>());
      }
      NdUtil::ReduceSum(ctx->stream(), Var({1, m}, gamma_diff->mut_dptr<float16>()),
                        Val({num_blocks, m}, tmp_gamma_diff), Var({num_blocks, m}, tmp_reduce_buf));
      NdUtil::ReduceSum(ctx->stream(), Var({1, m}, beta_diff->mut_dptr<float16>()),
                        Val({num_blocks, m}, tmp_beta_diff), Var({num_blocks, m}, tmp_reduce_buf));
    } else {
      if (has_beta_diff) {
        user_op::Tensor* reduce_buf = ctx->Tensor4ArgNameAndIndex("reduce_buf", 0);
        CHECK_EQ(m, beta_diff->shape().elem_cnt());
        CHECK_EQ(dy->shape().elem_cnt() % m, 0);
        const int64_t n = dy->shape().elem_cnt() / m;
        NdUtil::ReduceSum(ctx->stream(), Var({1, m}, beta_diff->mut_dptr<float16>()),
                          Val({n, m}, dy->dptr<float16>()),
                          Var({n, m}, reduce_buf->mut_dptr<float16>()));
      }
      if (has_gamma_diff) {
        const user_op::Tensor* normalized = ctx->Tensor4ArgNameAndIndex("normalized", 0);
        user_op::Tensor* reduce_buf = ctx->Tensor4ArgNameAndIndex("reduce_buf", 0);
        CHECK_EQ(m, gamma_diff->shape().elem_cnt());
        CHECK_EQ(dy->shape().elem_cnt() % m, 0);
        const int64_t n = dy->shape().elem_cnt() / m;
        NdUtil::BroadcastMul(ctx->stream(), Var({n, m}, reduce_buf->mut_dptr<float16>()),
                             Val({n, m}, normalized->dptr<float16>()),
                             Val({n, m}, dy->dptr<float16>()));
        NdUtil::ReduceSum(ctx->stream(), Var({1, m}, gamma_diff->mut_dptr<float16>()),
                          Val({n, m}, reduce_buf->dptr<float16>()),
                          Var({n, m}, reduce_buf->mut_dptr<float16>()));
      }
      if (has_normalized_diff) {
        if (has_gamma) {
          CHECK_EQ(m, gamma->shape().elem_cnt());
          CHECK_EQ(dy->shape().elem_cnt() % m, 0);
          const int64_t n = dy->shape().elem_cnt() / m;
          NdUtil::BroadcastMul(ctx->stream(), Var({n, m}, normalized_diff->mut_dptr<float16>()),
                               Val({n, m}, dy->dptr<float16>()),
                               Val({1, m}, gamma->dptr<float16>()));
        } else {
          Memcpy<DeviceType::kCUDA>(ctx->stream(), normalized_diff->mut_dptr<void>(),
                                    dy->dptr<void>(),
                                    dy->shape().elem_cnt() * GetSizeOfDataType(dy->data_type()));
        }
      }
    }
  }
};

REGISTER_USER_KERNEL("layer_norm_param_grad")
    .SetCreateFn<LayerNormParamGradGpuHalfKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)
                     && (user_op::HobDataType("dy", 0) == DataType::kFloat16))
    .SetInferTmpSizeFn([](user_op::InferContext* ctx) {
      const int64_t begin_params_axis = ctx->Attr<int64_t>("begin_params_axis");
      const bool has_gamma_diff = ctx->has_output("gamma_diff", 0);
      const bool has_beta_diff = ctx->has_output("beta_diff", 0);
      const bool has_normalized_diff = ctx->has_output("normalized_diff", 0);
      const auto& dy = ctx->InputTensorDesc("dy", 0);
      const int64_t instance_size = dy.shape().Count(begin_params_axis);
      size_t tmp_buffer_size = 0;
      if (has_gamma_diff && has_beta_diff && has_normalized_diff) {
        const size_t tmp_gamma_diff =
            GetCudaAlignedSize(GetLayerNormParamGradNumBlocks(dy.shape().elem_cnt()) * instance_size
                               * sizeof(float16));
        const size_t tmp_beta_diff = tmp_gamma_diff;
        const size_t tmp_reduce_buf = tmp_gamma_diff;
        tmp_buffer_size = tmp_gamma_diff + tmp_beta_diff + tmp_reduce_buf;
      } else {
        tmp_buffer_size = 0;
      }
      return tmp_buffer_size;
    });
}  // namespace oneflow
