#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/framework/random_generator.h"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/common/container_util.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/user/kernels/op_kernel_wrapper.h"
#include "oneflow/user/kernels/arange_kernel_util.h"
#include "oneflow/user/kernels/radix_sort.cuh"
#include "oneflow/user/kernels/random_seed_util.h"
#include "oneflow/user/kernels/distributions/common.h"
#include "oneflow/user/kernels/distributions/distribution_template_util.cuh"
#include "oneflow/core/ep/include/device.h"
#include "oneflow/core/ep/include/stream.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/job/nd_sbp_util.h"
#include "oneflow/core/register/tensor_slice_view.h"

namespace oneflow {
__global__ void GeneKeysAndValues(const int32_t n, uint64_t seed, uint64_t offset, int32_t* values,
                                  int32_t* keys) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, id, offset, &state);
  CUDA_1D_KERNEL_LOOP(i, n) {
    keys[i] = hiprand(&state);
    values[i] = i;
  }
}

__global__ void tempcopy2output(const int32_t n, const int32_t offset, int32_t* temp,
                                int32_t* output) {
  CUDA_1D_KERNEL_LOOP(i, n) { output[i] = temp[offset + i]; }
}
class GpuRandPermKernelCache final : public user_op::OpKernelCache {
 public:
  GpuRandPermKernelCache(int32_t lower, int32_t upper) : lower_(lower), upper_(upper) {}
  ~GpuRandPermKernelCache() override = default;

  int32_t lower() const { return lower_; }
  int32_t upper() const { return upper_; }

 private:
  const int32_t lower_;
  const int32_t upper_;
};

namespace {

template<typename K>
size_t GetCubSortPairsTempStorageSize(int64_t n) {
  size_t cub_sort_temp_store_size = 0;
  OF_CUDA_CHECK((hipcub::DeviceRadixSort::SortPairs<K, K>(nullptr, cub_sort_temp_store_size, nullptr,
                                                       nullptr, nullptr, nullptr, n)));
  size_t temp_store_size = GetCudaAlignedSize(cub_sort_temp_store_size);
  CHECK_GE(temp_store_size, 0) << "temp_store_size should >= 0.";
  CHECK_LT(temp_store_size, static_cast<size_t>(GetMaxVal<int64_t>()))
      << "temp_store_size should < " << static_cast<size_t>(GetMaxVal<int64_t>());
  return temp_store_size;
}

}  // namespace

class GpuRandPermKernel final : public user_op::OpKernel {
 public:
  GpuRandPermKernel() = default;
  ~GpuRandPermKernel() = default;
  std::shared_ptr<user_op::OpKernelCache> InitOpKernelCache(
      user_op::KernelCacheContext* ctx) const override {
    int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    if (parallel_num > 1) {
      const NdSbp& nd_sbp = ctx->NdSbp4ArgNameAndIndex("out", 0);
      const Shape& hierarchy = *ctx->parallel_desc().hierarchy();
      int64_t parallel_id = ctx->parallel_ctx().parallel_id();
      int32_t n = ctx->Attr<int32_t>("n");
      const Shape& logical_shape = Shape({n});
      TensorSliceView view =
          GetTensorSliceView4ParallelId(hierarchy, nd_sbp, logical_shape, parallel_id);
      std::shared_ptr<GpuRandPermKernelCache> cache(
          new GpuRandPermKernelCache(view.At(0).begin(), view.At(0).end()));
      return cache;
    } else {
      return nullptr;
    }
  }
  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    const auto& generator = CHECK_JUST(one::MakeGenerator(kCUDA));
    generator->set_current_seed(
        CHECK_JUST(GetOpKernelRandomSeedInCurrentRank(ctx, ctx->Attr<int64_t>("seed"))));
    return std::make_shared<DistributionKernelState>(generator);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache* cache) const override {
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    int32_t* output = out->mut_dptr<int32_t>();
    const int32_t n = ctx->Attr<int32_t>("n");
    if (n == 0) { return; }
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    auto* distribution_state = dynamic_cast<DistributionKernelState*>(state);
    CHECK_NOTNULL(distribution_state);
    const auto& generator = distribution_state->generator();
    CHECK_NOTNULL(generator);
    auto* stream = ctx->stream();
    const auto device_index = stream->device()->device_index();
    const auto& gpu_generator = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>(device_index));

    ep::CudaStream* hip_stream = stream->As<ep::CudaStream>();
    auto execution_policy = gpu_generator->CalcExecutionPolicy(n, hip_stream);

    auto counter_offset = std::get<0>(execution_policy);
    auto grid = std::get<1>(execution_policy);
    auto block = std::get<2>(execution_policy);

    uint64_t seed = gpu_generator->current_seed();
    uint64_t offset = gpu_generator->get_philox_offset(counter_offset);

    // layout for tmp |...key(in and out,2xN)..|....value....|.... space for sort function....|
    // values are the desired indexes ,and keys are generated randomly.
    void* tmp = tmp_buffer->mut_dptr<void>();
    int32_t* key_base = reinterpret_cast<int32_t*>(tmp);

    const int32_t key_aligned_bytes = GetCudaAlignedSize(n * sizeof(int32_t));
    int32_t* value_base =
        reinterpret_cast<int32_t*>(reinterpret_cast<char*>(key_base) + 2 * key_aligned_bytes);
    const int32_t indices_aligned_bytes = GetCudaAlignedSize(n * sizeof(int32_t));
    int32_t* temp_buffer_base =
        reinterpret_cast<int32_t*>(reinterpret_cast<char*>(value_base) + indices_aligned_bytes);
    const int32_t temp_buffer_aligned_bytes = GetCudaAlignedSize(n * sizeof(int32_t));

    void* tmp_base = reinterpret_cast<void*>(reinterpret_cast<char*>(temp_buffer_base)
                                             + temp_buffer_aligned_bytes);
    size_t temp_storage_bytes = GetCubSortPairsTempStorageSize<int32_t>(n);
    GeneKeysAndValues<<<grid, block, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
        n, seed, offset, value_base, key_base);
    if (cache == nullptr) {
      auto err = hipcub::DeviceRadixSort::SortPairs(
          /* d_temp_storage */ tmp_base,
          /* temp_storage_bytes */ temp_storage_bytes,
          /* d_keys_in */ key_base,
          /* d_keys_out */ key_base + n,
          /* d_values_in */ value_base,
          /* d_values_out */ output,
          /* num_items */ n,
          /* begin_bit */ 0,
          /* end_bit */ sizeof(int32_t) * 8,
          /* stream */ ctx->stream()->As<ep::CudaStream>()->hip_stream());
      OF_CUDA_CHECK(err);
    } else {
      auto err = hipcub::DeviceRadixSort::SortPairs(
          /* d_temp_storage */ tmp_base,
          /* temp_storage_bytes */ temp_storage_bytes,
          /* d_keys_in */ key_base,
          /* d_keys_out */ key_base + n,
          /* d_values_in */ value_base,
          /* d_values_out */ temp_buffer_base,
          /* num_items */ n,
          /* begin_bit */ 0,
          /* end_bit */ sizeof(int32_t) * 8,
          /* stream */ ctx->stream()->As<ep::CudaStream>()->hip_stream());
      OF_CUDA_CHECK(err);
      const auto* randperm_cache = dynamic_cast<const GpuRandPermKernelCache*>(cache);
      auto len = randperm_cache->upper() - randperm_cache->lower();
      const int64_t offset = randperm_cache->lower();
      int32_t block_num = gpu_generator->max_block_num();
      tempcopy2output<<<block_num, kCudaThreadsNumPerBlock, 0,
                        ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          len, offset, temp_buffer_base, output);
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};
REGISTER_USER_KERNEL("randperm")
    .SetCreateFn<GpuRandPermKernel>()
    .SetIsMatchedHob(user_op::HobDeviceType() == DeviceType::kCUDA)
    .SetInferTmpSizeFn([](user_op::InferContext* ctx) {
      const int32_t n = ctx->Attr<int32_t>("n");
      /* Sorted In */
      const int32_t sorted_in_aligned_bytes = 2 * GetCudaAlignedSize(n * sizeof(int32_t));
      /* Indices */
      const int32_t indices_aligned_bytes = GetCudaAlignedSize(n * sizeof(int32_t));
      const int32_t temp_aligned_bytes = GetCudaAlignedSize(n * sizeof(int32_t));

      /* CUB Temp Storage */
      const int32_t temp_storage_bytes = GetCubSortPairsTempStorageSize<int32_t>(n);

      return sorted_in_aligned_bytes + indices_aligned_bytes + temp_storage_bytes
             + temp_aligned_bytes;
    });

}  // namespace oneflow
