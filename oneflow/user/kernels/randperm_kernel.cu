#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "oneflow/core/common/data_type.h"
#include "oneflow/core/ep/include/stream.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/framework/random_generator.h"
#include "oneflow/user/kernels/op_kernel_wrapper.h"
#include "oneflow/user/kernels/arange_kernel_util.h"
#include "oneflow/user/kernels/radix_sort.cuh"
#include "oneflow/user/kernels/distributions/common.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {
__global__ void GeneKeysAndValues(const int32_t n, int32_t* values, int32_t* keys,
                                  hiprandState* state) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState local_state = state[id];
  CUDA_1D_KERNEL_LOOP(i, n) {
    keys[i] = hiprand(&local_state);
    values[i] = i;
  }
  state[id] = local_state;
}

namespace {

template<typename K>
size_t GetCubSortPairsTempStorageSize(int64_t n) {
  size_t cub_sort_temp_store_size = 0;
  OF_CUDA_CHECK((hipcub::DeviceRadixSort::SortPairs<K, K>(nullptr, cub_sort_temp_store_size, nullptr,
                                                       nullptr, nullptr, nullptr, n)));
  size_t temp_store_size = GetCudaAlignedSize(cub_sort_temp_store_size);
  CHECK_GE(temp_store_size, 0) << "temp_store_size should >= 0.";
  CHECK_LT(temp_store_size, static_cast<size_t>(GetMaxVal<int64_t>()))
      << "temp_store_size should < " << static_cast<size_t>(GetMaxVal<int64_t>());
  return temp_store_size;
}

}  // namespace

class GpuRandPermKernel final : public user_op::OpKernel {
 public:
  GpuRandPermKernel() = default;
  ~GpuRandPermKernel() = default;
  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    const auto& generator = CHECK_JUST(one::MakeGenerator(kCUDA));
    generator->set_current_seed(ctx->Attr<int64_t>("seed"));
    return std::make_shared<DistributionKernelState>(generator);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    int32_t* output = out->mut_dptr<int32_t>();
    const int32_t n = ctx->Attr<int32_t>("n");
    if (n == 0) { return; }
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    auto* distribution_state = dynamic_cast<DistributionKernelState*>(state);
    CHECK_NOTNULL(distribution_state);
    const auto& generator = distribution_state->generator();
    const auto& gpu_generator = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>());
    CHECK_NOTNULL(generator);

    int32_t block_num = gpu_generator->max_block_num();
    int32_t thread_num = gpu_generator->max_thread_num();
    hiprandState* curand_states = gpu_generator->curand_states();

    // layout for tmp |...key(in and out,2xN)..|....value....|.... space for sort function....|
    // values are the desired indexes ,and keys are generated randomly.
    void* tmp = tmp_buffer->mut_dptr<void>();
    int32_t* key_base = reinterpret_cast<int32_t*>(tmp);

    const int32_t key_aligned_bytes = GetCudaAlignedSize(n * sizeof(int32_t));
    int32_t* value_base =
        reinterpret_cast<int32_t*>(reinterpret_cast<char*>(key_base) + 2 * key_aligned_bytes);

    const int32_t indices_aligned_bytes = GetCudaAlignedSize(n * sizeof(int32_t));
    void* tmp_base =
        reinterpret_cast<void*>(reinterpret_cast<char*>(value_base) + indices_aligned_bytes);
    size_t temp_storage_bytes = GetCubSortPairsTempStorageSize<int32_t>(n);

    GeneKeysAndValues<<<block_num, thread_num, 0,
                        ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        n, value_base, key_base, curand_states);

    auto err = hipcub::DeviceRadixSort::SortPairs(
        /* d_temp_storage */ tmp_base,
        /* temp_storage_bytes */ temp_storage_bytes,
        /* d_keys_in */ key_base,
        /* d_keys_out */ key_base + n,
        /* d_values_in */ value_base,
        /* d_values_out */ output,
        /* num_items */ n,
        /* begin_bit */ 0,
        /* end_bit */ sizeof(int32_t) * 8,
        /* stream */ ctx->stream()->As<ep::CudaStream>()->hip_stream());
    OF_CUDA_CHECK(err);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};
REGISTER_USER_KERNEL("randperm")
    .SetCreateFn<GpuRandPermKernel>()
    .SetIsMatchedHob(user_op::HobDeviceType() == DeviceType::kCUDA)
    .SetInferTmpSizeFn([](user_op::InferContext* ctx) {
      const int32_t n = ctx->Attr<int32_t>("n");
      /* Sorted In */
      const int32_t sorted_in_aligned_bytes = 2 * GetCudaAlignedSize(n * sizeof(int32_t));
      /* Indices */
      const int32_t indices_aligned_bytes = GetCudaAlignedSize(n * sizeof(int32_t));

      /* CUB Temp Storage */
      const int32_t temp_storage_bytes = GetCubSortPairsTempStorageSize<int32_t>(n);

      return sorted_in_aligned_bytes + indices_aligned_bytes + temp_storage_bytes;
    });

}  // namespace oneflow
