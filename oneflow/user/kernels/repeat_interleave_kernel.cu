#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/roll_kernel_utils.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

#include <algorithm>

namespace oneflow {

namespace {

template<typename T>
__global__ void repeat_interleave(const T* in_ptr, const T* cumsum_ptr, T* out_ptr,
                                  const int64_t num) {
  CUDA_1D_KERNEL_LOOP(i, num) {
    T end = cumsum_ptr[i];
    T size = in_ptr[i];
    T start = end - size;
    for (T j = start; j < end; j++) { out_ptr[j] = i; }
  }
}

}  // namespace

template<typename T>
class GpuRepeatInterLeaveKernel final : public user_op::OpKernel {
 public:
  GpuRepeatInterLeaveKernel() = default;
  ~GpuRepeatInterLeaveKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    const user_op::Tensor* cumsum = ctx->Tensor4ArgNameAndIndex("cumsum", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int64_t& repeat_num = ctx->Attr<std::int64_t>("repeat_num");
    const T* in_ptr = in->dptr<T>();
    const T* cumsum_ptr = cumsum->dptr<T>();
    T* out_ptr = out->mut_dptr<T>();

    repeat_interleave<T><<<BlocksNum4ThreadsNum(in->shape_view().At(0)), kCudaThreadsNumPerBlock, 0,
                           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        in_ptr, cumsum_ptr, out_ptr, in->shape_view().At(0));
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_REPEAT_INTER_LEAVE_KERNEL(dtype)                      \
  REGISTER_USER_KERNEL("repeat_interleave")                            \
      .SetCreateFn<GpuRepeatInterLeaveKernel<dtype>>()                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("in", 0) == GetDataType<dtype>::value))

REGISTER_REPEAT_INTER_LEAVE_KERNEL(int32_t);
REGISTER_REPEAT_INTER_LEAVE_KERNEL(int64_t);

}  // namespace oneflow
