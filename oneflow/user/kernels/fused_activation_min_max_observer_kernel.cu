#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include "oneflow/core/ndarray/binary_func.h"
#include "oneflow/core/kernel/util/numeric_limits.cuh"

namespace oneflow {

namespace {

template<int M>
__host__ __device__ int ModDiv(int64_t N) {
  return N - (N / M * M);
}

template<>
__host__ __device__ int ModDiv<2>(int64_t N) {
  return N & 0x1;
}

template<>
__host__ __device__ int ModDiv<4>(int64_t N) {
  return N & 0x3;
}

template<>
__host__ __device__ int ModDiv<8>(int64_t N) {
  return N & 0x7;
}

template<>
__host__ __device__ int ModDiv<16>(int64_t N) {
  return N & 0xF;
}

template<int pack_size, typename T>
__global__ void ReduceMinMaxPerLayer(const int64_t elements, const T* in_ptr, T* min_max_ptr) {
  using LoadType = cuda::elementwise::PackType<T, pack_size>;
  using LoadPack = cuda::elementwise::Pack<T, pack_size>;
  using MinMaxPack = cuda::elementwise::Pack<T, 2>;

  extern __shared__ uint8_t buffer[];

  MinMaxPack min_max;
  min_max.elem[0] = detail::numeric_limits<T>::max();
  min_max.elem[1] = detail::numeric_limits<T>::lowest();

  int64_t gid = (blockDim.x * blockIdx.x) + threadIdx.x;
  int64_t step = gridDim.x * blockDim.x * pack_size;

  for (int64_t idx = gid * pack_size; idx < elements; idx += step) {
    LoadPack in;
    in.storage = reinterpret_cast<const LoadType*>(in_ptr + idx)[0];
#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      min_max.elem[0] = BinaryFuncMin<T>::Invoke(min_max.elem[0], in.elem[i]);
      min_max.elem[1] = BinaryFuncMax<T>::Invoke(min_max.elem[1], in.elem[i]);
    }
  }
  int rest = ModDiv<pack_size>(elements);
  if (rest > 0 && gid == (gridDim.x * blockDim.x - 1)) {
    in_ptr += elements - rest;
    LoadPack in;
    in.storage = reinterpret_cast<const LoadType*>(in_ptr)[0];
#pragma unroll
    for (int i = 0; i < rest; ++i) {
      min_max.elem[0] = BinaryFuncMin<T>::Invoke(min_max.elem[0], in.elem[i]);
      min_max.elem[1] = BinaryFuncMax<T>::Invoke(min_max.elem[1], in.elem[i]);
    }
  }

  int64_t tid = threadIdx.x;

  MinMaxPack* shared_min_max = reinterpret_cast<MinMaxPack*>(buffer);
  shared_min_max[tid].storage = min_max.storage;
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      MinMaxPack min_max0, min_max1;
      min_max0.storage = shared_min_max[tid].storage;
      min_max1.storage = shared_min_max[tid + s].storage;
      min_max0.elem[0] = BinaryFuncMin<T>::Invoke(min_max0.elem[0], min_max1.elem[0]);
      min_max0.elem[1] = BinaryFuncMax<T>::Invoke(min_max0.elem[1], min_max1.elem[1]);
      shared_min_max[tid].storage = min_max0.storage;
    }
    __syncthreads();
  }

  if (tid == 0) {
    reinterpret_cast<MinMaxPack*>(min_max_ptr)[blockIdx.x].storage = shared_min_max[0].storage;
  }
}

template<typename T, typename Q>
__global__ void ComputeOFScaleAndZeroPoint(const T* min_max_ptr, const int min_max_size,
                                           const int quantization_bit, const float* weight_scale,
                                           const float* weight_acc, const T* bias, T* in_scale,
                                           Q* in_zero_point, T* out_scale, T* out_bias,
                                           const int out_elements) {
  using MinMaxPack = cuda::elementwise::Pack<T, 2>;

  extern __shared__ uint8_t buffer[];
  MinMaxPack* shared_min_max = reinterpret_cast<MinMaxPack*>(buffer);
  int64_t tid = threadIdx.x;
  {
    MinMaxPack min_max;
    min_max.elem[0] = detail::numeric_limits<T>::max();
    min_max.elem[1] = detail::numeric_limits<T>::lowest();
#pragma unroll
    for (int64_t idx = threadIdx.x; idx < min_max_size; idx += blockDim.x) {
      MinMaxPack in = reinterpret_cast<const MinMaxPack*>(min_max_ptr)[idx];
      min_max.elem[0] = BinaryFuncMin<T>::Invoke(min_max.elem[0], in.elem[0]);
      min_max.elem[1] = BinaryFuncMax<T>::Invoke(min_max.elem[1], in.elem[1]);
    }
    shared_min_max[tid].storage = min_max.storage;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
      if (tid < s) {
        MinMaxPack min_max0, min_max1;
        min_max0.storage = shared_min_max[tid].storage;
        min_max1.storage = shared_min_max[tid + s].storage;
        min_max0.elem[0] = BinaryFuncMin<T>::Invoke(min_max0.elem[0], min_max1.elem[0]);
        min_max0.elem[1] = BinaryFuncMax<T>::Invoke(min_max0.elem[1], min_max1.elem[1]);
        shared_min_max[tid].storage = min_max0.storage;
      }
      __syncthreads();
    }
  }

  MinMaxPack min_max = shared_min_max[0];
  float min_value = static_cast<float>(min_max.elem[0]);
  float max_value = static_cast<float>(min_max.elem[1]);
  float input_scale = (max_value - min_value) / ((1 << quantization_bit) - 1);
  int32_t input_zero_point =
      -(__float2int_rn(min_value / input_scale) + (1 << (quantization_bit - 1)));
  float scale_zero_point = -input_scale * input_zero_point;

  int64_t thread_num = gridDim.x * blockDim.x;
  int64_t gid = (blockDim.x * blockIdx.x) + threadIdx.x;
  if (gid == 0) {
    in_scale[0] = static_cast<T>(input_scale);
    in_zero_point[0] = static_cast<Q>(input_zero_point);
  }

  using LoadWPack = cuda::elementwise::Pack<float, 4>;
  using LoadBPack = cuda::elementwise::Pack<T, 4>;
  using StorePack = cuda::elementwise::Pack<T, 4>;

  if (bias) {
    for (int64_t idx = gid << 2; idx < out_elements; idx += thread_num << 2) {
      LoadWPack w_scale = reinterpret_cast<const LoadWPack*>(weight_scale + idx)[0];
      LoadWPack w_acc = reinterpret_cast<const LoadWPack*>(weight_acc + idx)[0];
      LoadBPack b = reinterpret_cast<const LoadBPack*>(bias + idx)[0];
      StorePack store_scale, store_bias;

      store_scale.elem[0] = static_cast<T>(w_scale.elem[0] * input_scale);
      store_scale.elem[1] = static_cast<T>(w_scale.elem[1] * input_scale);
      store_scale.elem[2] = static_cast<T>(w_scale.elem[2] * input_scale);
      store_scale.elem[3] = static_cast<T>(w_scale.elem[3] * input_scale);

      store_bias.elem[0] = static_cast<T>(__fmaf_rn(w_acc.elem[0], scale_zero_point, b.elem[0]));
      store_bias.elem[1] = static_cast<T>(__fmaf_rn(w_acc.elem[1], scale_zero_point, b.elem[1]));
      store_bias.elem[2] = static_cast<T>(__fmaf_rn(w_acc.elem[2], scale_zero_point, b.elem[2]));
      store_bias.elem[3] = static_cast<T>(__fmaf_rn(w_acc.elem[3], scale_zero_point, b.elem[3]));

      reinterpret_cast<StorePack*>(out_scale + idx)[0] = store_scale;
      reinterpret_cast<StorePack*>(out_bias + idx)[0] = store_bias;
    }
    int rest = ModDiv<4>(out_elements);
    if (rest > 0 && gid == (thread_num - 1)) {
      int offset = out_elements - rest;
      LoadWPack w_scale = reinterpret_cast<const LoadWPack*>(weight_scale + offset)[0];
      LoadWPack w_acc = reinterpret_cast<const LoadWPack*>(weight_acc + offset)[0];
      LoadBPack b = reinterpret_cast<const LoadBPack*>(bias + offset)[0];
#pragma unroll
      for (int i = 0; i < rest; ++i) {
        out_scale[offset + i] = static_cast<T>(w_scale.elem[i] * input_scale);
        out_bias[offset + i] =
            static_cast<T>(__fmaf_rn(w_acc.elem[i], scale_zero_point, b.elem[i]));
      }
    }
  } else {
    for (int64_t idx = gid << 2; idx < out_elements; idx += thread_num << 2) {
      LoadWPack w_scale = reinterpret_cast<const LoadWPack*>(weight_scale + idx)[0];
      LoadWPack w_acc = reinterpret_cast<const LoadWPack*>(weight_acc + idx)[0];
      StorePack store_scale, store_bias;

      store_scale.elem[0] = static_cast<T>(w_scale.elem[0] * input_scale);
      store_scale.elem[1] = static_cast<T>(w_scale.elem[1] * input_scale);
      store_scale.elem[2] = static_cast<T>(w_scale.elem[2] * input_scale);
      store_scale.elem[3] = static_cast<T>(w_scale.elem[3] * input_scale);

      store_bias.elem[0] = static_cast<T>(w_acc.elem[0] * scale_zero_point);
      store_bias.elem[1] = static_cast<T>(w_acc.elem[1] * scale_zero_point);
      store_bias.elem[2] = static_cast<T>(w_acc.elem[2] * scale_zero_point);
      store_bias.elem[3] = static_cast<T>(w_acc.elem[3] * scale_zero_point);

      reinterpret_cast<StorePack*>(out_scale + idx)[0] = store_scale;
      reinterpret_cast<StorePack*>(out_bias + idx)[0] = store_bias;
    }
    int rest = ModDiv<4>(out_elements);
    if (rest > 0 && gid == (thread_num - 1)) {
      int offset = out_elements - rest;
      LoadWPack w_scale = reinterpret_cast<const LoadWPack*>(weight_scale + offset)[0];
      LoadWPack w_acc = reinterpret_cast<const LoadWPack*>(weight_acc + offset)[0];
#pragma unroll
      for (int i = 0; i < rest; ++i) {
        out_scale[offset + i] = static_cast<T>(w_scale.elem[i] * input_scale);
        out_bias[offset + i] = static_cast<T>(w_acc.elem[i] * scale_zero_point);
      }
    }
  }
}

}  // namespace

template<typename T>
class GpuFusedActivationMinMaxObserverKernel final : public user_op::OpKernel {
 public:
  GpuFusedActivationMinMaxObserverKernel() = default;
  ~GpuFusedActivationMinMaxObserverKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    const user_op::Tensor* weight_scale = ctx->Tensor4ArgNameAndIndex("weight_scale", 0);
    const user_op::Tensor* weight_acc = ctx->Tensor4ArgNameAndIndex("weight_acc", 0);
    const user_op::Tensor* bias = nullptr;
    if (ctx->has_input("bias", 0)) { bias = ctx->Tensor4ArgNameAndIndex("bias", 0); }

    user_op::Tensor* in_scale = ctx->Tensor4ArgNameAndIndex("in_scale", 0);
    user_op::Tensor* in_zero_point = ctx->Tensor4ArgNameAndIndex("in_zero_point", 0);
    user_op::Tensor* out_scale = ctx->Tensor4ArgNameAndIndex("out_scale", 0);
    user_op::Tensor* out_bias = ctx->Tensor4ArgNameAndIndex("out_bias", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    const std::string quantization_scheme = ctx->Attr<std::string>("quantization_scheme");
    const int32_t quantization_bit = ctx->Attr<int32_t>("quantization_bit");
    const bool per_layer_quantization = ctx->Attr<bool>("per_layer_quantization");
    const std::string quantization_formula = ctx->Attr<std::string>("quantization_formula");

    CHECK(quantization_scheme == "affine");
    CHECK(quantization_bit == 8);

    const int64_t elements = in->shape_view().elem_cnt();

    constexpr int pack_size = cuda::elementwise::PackSize<T>();
    int grid_size = 0;
    int64_t pack_num = (elements + pack_size - 1) / pack_size;
    cuda::elementwise::GetNumBlocks(pack_num, &grid_size);
    grid_size = grid_size > 2048 ? 2048 : grid_size;

    size_t element_bytes = GetSizeOfDataType(GetDataType<T>::value);
    CHECK_GE(tmp_buffer->shape_view().elem_cnt(), grid_size * element_bytes * 2);

    T* min_max = reinterpret_cast<T*>(tmp_buffer->mut_dptr());
    auto stream = ctx->stream()->As<ep::CudaStream>()->hip_stream();
    if (per_layer_quantization) {
      ReduceMinMaxPerLayer<pack_size, T>
          <<<grid_size, cuda::elementwise::kBlockSize,
             cuda::elementwise::kBlockSize * element_bytes * 2, stream>>>(elements, in->dptr<T>(),
                                                                          min_max);
    } else {
      UNIMPLEMENTED()
          << "fused_activation_min_max_observer does not support per-channel quantization";
    }

    if (quantization_formula == "oneflow") {
      if (quantization_bit == 8) {
        ComputeOFScaleAndZeroPoint<T, int8_t>
            <<<1, cuda::elementwise::kBlockSize, cuda::elementwise::kBlockSize * element_bytes * 2,
               stream>>>(min_max, grid_size, quantization_bit, weight_scale->dptr<float>(),
                         weight_acc->dptr<float>(), bias ? bias->dptr<T>() : nullptr,
                         in_scale->mut_dptr<T>(), in_zero_point->mut_dptr<int8_t>(),
                         out_scale->mut_dptr<T>(), out_bias->mut_dptr<T>(),
                         out_scale->shape_view().elem_cnt());
      } else {
        UNIMPLEMENTED();
      }
    } else {
      UNIMPLEMENTED()
          << "fused_activation_min_max_observer only support oneflow quantization formula";
    }
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_ACTIVATION_MIN_MAX_OBSERVER_KERNEL(dtype)                        \
  REGISTER_USER_KERNEL("fused_activation_min_max_observer")                             \
      .SetCreateFn<GpuFusedActivationMinMaxObserverKernel<dtype>>()                     \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("in", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) -> size_t { return 128 * 1024 * 1024; })

REGISTER_FUSED_ACTIVATION_MIN_MAX_OBSERVER_KERNEL(float);
REGISTER_FUSED_ACTIVATION_MIN_MAX_OBSERVER_KERNEL(double);
REGISTER_FUSED_ACTIVATION_MIN_MAX_OBSERVER_KERNEL(half);

}  // namespace oneflow
