#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/user/kernels/roll_kernel_utils.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void RollCudaKernel(const T* in_ptr, const SHIFTS shifts, const SHAPE shape,
                               const STRIDE stride, const int32_t dims, const int64_t elements,
                               T* out_ptr) {
  int32_t gid = (blockDim.x * blockIdx.x) + threadIdx.x;
  int32_t step = gridDim.x * blockDim.x;
  while (gid < elements) {
    int32_t offset = getShiftedOffset(gid, shifts.val, shape.val, stride.val, dims);
    out_ptr[gid] = in_ptr[offset];
    gid += step;
  }
}

template<typename T>
struct GpuRollFunctor final {
  void operator()(DeviceCtx* ctx, const T* in_ptr, const SHIFTS shifts, const SHAPE shape,
                  const STRIDE stride, const int32_t dims, const int64_t elements, T* out_ptr) {
    RollCudaKernel<T>
        <<<BlocksNum4ThreadsNum(elements), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            in_ptr, shifts, shape, stride, dims, elements, out_ptr);
  }
};

template<>
void GpuRollFunctor<float16>::operator()(DeviceCtx* ctx, const float16* in_ptr, const SHIFTS shifts,
                                         const SHAPE shape, const STRIDE stride, const int32_t dims,
                                         const int64_t elements, float16* out_ptr) {
  RollCudaKernel<half>
      <<<BlocksNum4ThreadsNum(elements), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          reinterpret_cast<const half*>(in_ptr), shifts, shape, stride, dims, elements,
          reinterpret_cast<half*>(out_ptr));
}

template<typename T>
__global__ void RollFlattenCudaKernel(const T* in_ptr, const int32_t shifts, const int64_t elements,
                                      T* out_ptr) {
  int32_t gid = (blockDim.x * blockIdx.x) + threadIdx.x;
  int32_t step = gridDim.x * blockDim.x;

  while (gid < elements) {
    int32_t shifted_idx = (gid - shifts) % elements;
    if (shifted_idx < 0) shifted_idx = shifted_idx + elements;
    out_ptr[gid] = in_ptr[shifted_idx];
    gid += step;
  }
}

template<typename T>
struct GpuRollFlattenFunctor final {
  void operator()(DeviceCtx* ctx, const T* in_ptr, const int32_t shifts, const int64_t elements,
                  T* out_ptr) {
    RollFlattenCudaKernel<T>
        <<<BlocksNum4ThreadsNum(elements), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            in_ptr, shifts, elements, out_ptr);
  }
};

template<>
void GpuRollFlattenFunctor<float16>::operator()(DeviceCtx* ctx, const float16* in_ptr,
                                                const int32_t shifts, const int64_t elements,
                                                float16* out_ptr) {
  RollFlattenCudaKernel<half>
      <<<BlocksNum4ThreadsNum(elements), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          reinterpret_cast<const half*>(in_ptr), shifts, elements,
          reinterpret_cast<half*>(out_ptr));
}

template<typename T>
__global__ void Roll1DimCudaKernel(const T* in_ptr, const int32_t start, const int32_t size,
                                   const int32_t stride, const int64_t elements, T* out_ptr) {
  int32_t gid = (blockDim.x * blockIdx.x) + threadIdx.x;
  int32_t step = gridDim.x * blockDim.x;

  while (gid < elements) {
    // roll dim idx is the index of linear_index along the rolling dimension.
    int32_t roll_dim_idx = gid % (stride * size) / stride;
    // index into the source data to find appropriate value.
    int32_t source_idx = 0;
    if (roll_dim_idx >= (size - start)) {
      source_idx = gid - ((size - start) * stride);
    } else {
      source_idx = gid + (start * stride);
    }
    out_ptr[gid] = in_ptr[source_idx];

    gid += step;
  }
}

template<typename T>
struct GpuRoll1DimFunctor final {
  void operator()(DeviceCtx* ctx, const T* in_ptr, const int32_t start, const int32_t size,
                  const int32_t stride, const int64_t elements, T* out_ptr) {
    Roll1DimCudaKernel<T>
        <<<BlocksNum4ThreadsNum(elements), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            in_ptr, start, size, stride, elements, out_ptr);
  }
};

template<>
void GpuRoll1DimFunctor<float16>::operator()(DeviceCtx* ctx, const float16* in_ptr,
                                             const int32_t start, const int32_t size,
                                             const int32_t stride, const int64_t elements,
                                             float16* out_ptr) {
  Roll1DimCudaKernel<half>
      <<<BlocksNum4ThreadsNum(elements), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          reinterpret_cast<const half*>(in_ptr), start, size, stride, elements,
          reinterpret_cast<half*>(out_ptr));
}

}  // namespace

template<typename T>
class GpuRollKernel final : public user_op::OpKernel {
 public:
  GpuRollKernel() = default;
  ~GpuRollKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const std::vector<int32_t>& shifts = ctx->Attr<std::vector<int32_t>>("shifts");
    const std::vector<int32_t>& dims = ctx->Attr<std::vector<int32_t>>("dims");

    const T* in_ptr = in->dptr<T>();
    T* out_ptr = out->mut_dptr<T>();
    const int64_t elem_count = out->shape().elem_cnt();

    if (dims[0] == -1) {
      GpuRollFlattenFunctor<T>()(ctx->device_ctx(), in_ptr, shifts[0], elem_count, out_ptr);
    } else {
      SHAPE new_shape{};
      SHIFTS new_shifts{};
      int32_t num_axes = 0;
      computeParams(in->shape(), shifts, dims, new_shifts.val, new_shape.val, &num_axes);

      STRIDE stride{};
      initStride(stride, new_shape, num_axes);

      if (dims.size() == 1) {
        // NOTE(Liang Depeng): borrow the implementation of pytorch
        const int32_t size = new_shape.val[dims[0]];
        int32_t start = (size - new_shifts.val[dims[0]]) % size;
        // Behavior of % is different in C++ vs Python for negative numbers. This
        // corrects the difference.
        if (start < 0) start = start + size;

        GpuRoll1DimFunctor<T>()(ctx->device_ctx(), in_ptr, start, size, stride.val[dims[0]],
                                elem_count, out_ptr);

      } else {
        GpuRollFunctor<T>()(ctx->device_ctx(), in_ptr, new_shifts, new_shape, stride, num_axes,
                            elem_count, out_ptr);
      }
    }
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_ROLL_KERNEL(dtype)                                                 \
  REGISTER_USER_KERNEL("roll").SetCreateFn<GpuRollKernel<dtype>>().SetIsMatchedHob( \
      (user_op::HobDeviceTag() == DeviceType::kGPU)                                 \
      & (user_op::HobDataType("in", 0) == GetDataType<dtype>::value))

REGISTER_ROLL_KERNEL(float);
REGISTER_ROLL_KERNEL(double);
REGISTER_ROLL_KERNEL(float16);
REGISTER_ROLL_KERNEL(uint8_t);
REGISTER_ROLL_KERNEL(int8_t);
REGISTER_ROLL_KERNEL(int32_t);
REGISTER_ROLL_KERNEL(int64_t);

}  // namespace oneflow
