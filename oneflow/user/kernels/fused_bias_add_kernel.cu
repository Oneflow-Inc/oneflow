#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/framework.h"

namespace oneflow {

namespace {

template<typename T>
struct GeluFunctor {
  __device__ T Compute(T x, int64_t i) const {
    return static_cast<T>(0.5) * x * (static_cast<T>(1.0) + erf(static_cast<T>(M_SQRT1_2) * x));
  }
};

template<>
struct GeluFunctor<half> {
  GeluFunctor<float> float_functor;
  __device__ half Compute(half x, int64_t i) const {
    return __float2half(float_functor.Compute(__half2float(x), i));
  }
  __device__ half2 ComputeHalf2(half2 x, int64_t i) const {
    half2 y;
    y.x = __float2half(float_functor.Compute(__half2float(x.x), 2 * i));
    y.y = __float2half(float_functor.Compute(__half2float(x.y), 2 * i + 1));
    return y;
  }
};

template<typename T>
struct MaskAndScaleFunctor {
  MaskAndScaleFunctor(const int8_t* mask, float scale) : mask(mask), scale(scale) {}
  __device__ T Compute(T x, int64_t i) const { return x * static_cast<T>(mask[i]) * scale; }
  const int8_t* mask;
  float scale;
};

template<>
struct MaskAndScaleFunctor<half> {
  MaskAndScaleFunctor(const int8_t* mask, float scale) : mask(mask), scale(scale) {}
  __device__ half Compute(half x, int64_t i) const {
    return x * static_cast<half>(mask[i] * scale);
  }
  __device__ half2 ComputeHalf2(half2 x, int64_t i) const {
    const char2* mask_c2 = reinterpret_cast<const char2*>(mask);
    char2 mask_val = mask_c2[i];
    half2 one_or_zero_h2;
    half2 h2_scale = __float2half2_rn(scale);
    one_or_zero_h2.x = mask_val.x;
    one_or_zero_h2.y = mask_val.y;
    return __hmul2(__hmul2(x, one_or_zero_h2), h2_scale);
  }
  const int8_t* mask;
  float scale;
};

template<typename T>
struct MaskAndScaleAddFunctor {
  MaskAndScaleAddFunctor(const int8_t* mask, const T* addend, float scale)
      : mask(mask), addend(addend), scale(scale) {}
  __device__ T Compute(T x, int64_t i) const {
    return x * static_cast<T>(mask[i]) * scale + addend[i];
  }
  const int8_t* mask;
  const T* addend;
  float scale;
};

template<>
struct MaskAndScaleAddFunctor<half> {
  MaskAndScaleAddFunctor(const int8_t* mask, const half* addend, float scale)
      : mask(mask), addend(addend), scale(scale) {}
  __device__ half Compute(half x, int64_t i) const {
    return x * static_cast<half>(mask[i] * scale) + addend[i];
  }
  __device__ half2 ComputeHalf2(half2 x, int64_t i) const {
    const char2* mask_c2 = reinterpret_cast<const char2*>(mask);
    const half2* addend_h2 = reinterpret_cast<const half2*>(addend);
    char2 mask_val = mask_c2[i];
    half2 one_or_zero_h2;
    half2 h2_scale = __float2half2_rn(scale);
    one_or_zero_h2.x = mask_val.x;
    one_or_zero_h2.y = mask_val.y;
    return __hadd2(__hmul2(__hmul2(x, one_or_zero_h2), h2_scale), addend_h2[i]);
  }
  const int8_t* mask;
  const half* addend;
  float scale;
};

template<typename T>
struct GeluGradFunctor {
  const T coef = sqrt(static_cast<T>(2.0) / acos(static_cast<T>(-1.0)));
  __device__ T Compute(T x, T dy, int64_t i) const {
    return static_cast<T>(0.5)
           * (static_cast<T>(1.0) + erf(static_cast<T>(M_SQRT1_2) * x)
              + x * coef * exp(static_cast<T>(-0.5) * x * x))
           * dy;
  }
};

template<>
struct GeluGradFunctor<half> {
  GeluGradFunctor<float> float_functor;
  __device__ half Compute(half x, half dy, int64_t i) const {
    return __float2half(float_functor.Compute(__half2float(x), __half2float(dy), i));
  }
};

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGpu(FUNCTOR functor, const Index elem_cnt, const Index bias_size,
                                const Index inner_size, const T* x, const T* bias, T* y) {
  const Index block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[(i % block_size) / inner_size];
    y[i] = functor.Compute(x_i, i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGradGpu(FUNCTOR grad_functor, const Index elem_cnt,
                                    const Index bias_size, const Index inner_size, const T* x,
                                    const T* bias, const T* dy, T* dx) {
  const Index block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[(i % block_size) / inner_size];
    dx[i] = grad_functor.Compute(x_i, dy[i], i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddRowGpu(FUNCTOR functor, const Index elem_cnt, const Index bias_size,
                                   const T* x, const T* bias, T* y) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i % bias_size];
    y[i] = functor.Compute(x_i, i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGradRowGpu(FUNCTOR grad_functor, const Index elem_cnt,
                                       const Index bias_size, const T* x, const T* bias,
                                       const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i % bias_size];
    dx[i] = grad_functor.Compute(x_i, dy[i], i);
  }
}

template<typename FUNCTOR, typename Index>
__global__ void FusedBiasAddRowGpuHalf2(FUNCTOR functor, const Index elem_cnt,
                                        const Index bias_size, const half* x, const half* bias,
                                        half* y) {
  const Index h2_elem_cnt = elem_cnt / 2;
  const Index h2_bias_size = bias_size / 2;
  const auto* x_h2 = reinterpret_cast<const half2*>(x);
  const auto* bias_h2 = reinterpret_cast<const half2*>(bias);
  auto* y_h2 = reinterpret_cast<half2*>(y);
  CUDA_1D_KERNEL_LOOP_T(Index, i, h2_elem_cnt) {
    half2 x_i = __hadd2(x_h2[i], bias_h2[i % h2_bias_size]);
    y_h2[i] = functor.ComputeHalf2(x_i, i);
  }
}

template<typename FUNCTOR, typename Index>
__global__ void FusedBiasAddGradRowGpuHalf2(FUNCTOR grad_functor, const Index elem_cnt,
                                            const Index bias_size, const half* x, const half* bias,
                                            const half* dy, half* dx) {
  const Index h2_elem_cnt = elem_cnt / 2;
  const Index h2_bias_size = bias_size / 2;
  const auto* x_h2 = reinterpret_cast<const half2*>(x);
  const auto* bias_h2 = reinterpret_cast<const half2*>(bias);
  const auto* dy_h2 = reinterpret_cast<const half2*>(dy);
  auto* dx_h2 = reinterpret_cast<half2*>(dx);
  CUDA_1D_KERNEL_LOOP_T(Index, i, h2_elem_cnt) {
    half2 x_i = __hadd2(x_h2[i], bias_h2[i % h2_bias_size]);
    half2 dy_i = dy_h2[i];
    half2 dx_i;
    dx_i.x = grad_functor.Compute(x_i.x, dy_i.x, 2 * i);
    dx_i.y = grad_functor.Compute(x_i.y, dy_i.y, 2 * i + 1);
    dx_h2[i] = dx_i;
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddColGpu(FUNCTOR functor, const Index elem_cnt, const Index inner_size,
                                   const T* x, const T* bias, T* y) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i / inner_size];
    y[i] = functor.Compute(x_i, i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGradColGpu(FUNCTOR grad_functor, const Index elem_cnt,
                                       const Index inner_size, const T* x, const T* bias,
                                       const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i / inner_size];
    dx[i] = grad_functor.Compute(x_i, dy[i], i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
struct FusedBiasAddRow {
  static void Invoke(DeviceCtx* ctx, FUNCTOR functor, Index elem_cnt, Index bias_size, const T* x,
                     const T* bias, T* y) {
    FusedBiasAddRowGpu<FUNCTOR, T, Index>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            functor, elem_cnt, bias_size, x, bias, y);
  }
};

template<typename FUNCTOR, typename Index>
struct FusedBiasAddRow<FUNCTOR, half, Index> {
  static void Invoke(DeviceCtx* ctx, FUNCTOR functor, Index elem_cnt, Index bias_size,
                     const half* x, const half* bias, half* y) {
    if (bias_size % 2 == 0) {
      FusedBiasAddRowGpuHalf2<FUNCTOR, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt / 2), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              functor, elem_cnt, bias_size, x, bias, y);
    } else {
      FusedBiasAddRowGpu<FUNCTOR, half, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              functor, elem_cnt, bias_size, x, bias, y);
    }
  }
};

template<typename FUNCTOR, typename T, typename Index>
void FusedBiasAddForwardImpl(DeviceCtx* ctx, FUNCTOR functor, Index outer_size, Index bias_size,
                             Index inner_size, const T* x, const T* bias, T* y) {
  const Index elem_cnt = outer_size * bias_size * inner_size;
  if (inner_size == 1) {
    FusedBiasAddRow<FUNCTOR, T, Index>::Invoke(ctx, functor, elem_cnt, bias_size, x, bias, y);
  } else if (outer_size == 1) {
    FusedBiasAddColGpu<FUNCTOR, T, Index>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            functor, elem_cnt, inner_size, x, bias, y);
  } else {
    FusedBiasAddGpu<FUNCTOR, T, Index>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            functor, elem_cnt, bias_size, inner_size, x, bias, y);
  }
}

template<typename FUNCTOR, typename T, typename Index>
struct FusedBiasAddGradRow {
  static void Invoke(DeviceCtx* ctx, FUNCTOR grad_functor, Index elem_cnt, Index bias_size,
                     const T* x, const T* bias, const T* dy, T* dx) {
    FusedBiasAddGradRowGpu<FUNCTOR, T, Index>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            grad_functor, elem_cnt, bias_size, x, bias, dy, dx);
  }
};

template<typename FUNCTOR, typename Index>
struct FusedBiasAddGradRow<FUNCTOR, half, Index> {
  static void Invoke(DeviceCtx* ctx, FUNCTOR grad_functor, Index elem_cnt, Index bias_size,
                     const half* x, const half* bias, const half* dy, half* dx) {
    if (bias_size % 2 == 0) {
      FusedBiasAddGradRowGpuHalf2<FUNCTOR, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt / 2), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              grad_functor, elem_cnt, bias_size, x, bias, dy, dx);
    } else {
      FusedBiasAddGradRowGpu<FUNCTOR, half, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              grad_functor, elem_cnt, bias_size, x, bias, dy, dx);
    }
  }
};

template<typename FUNCTOR, typename T, typename Index>
void FusedBiasAddGradImpl(DeviceCtx* ctx, FUNCTOR grad_functor, Index outer_size, Index bias_size,
                          Index inner_size, const T* x, const T* bias, const T* dy, T* dx) {
  const Index elem_cnt = outer_size * bias_size * inner_size;
  if (inner_size == 1) {
    FusedBiasAddGradRow<FUNCTOR, T, Index>::Invoke(ctx, grad_functor, elem_cnt, bias_size, x, bias,
                                                   dy, dx);
  } else if (outer_size == 1) {
    FusedBiasAddGradColGpu<FUNCTOR, T, Index>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            grad_functor, elem_cnt, inner_size, x, bias, dy, dx);
  } else {
    FusedBiasAddGradGpu<FUNCTOR, T, Index>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            grad_functor, elem_cnt, bias_size, inner_size, x, bias, dy, dx);
  }
}

template<typename FUNCTOR, typename T>
void DispatchFusedBiasAddForwardImpl(DeviceCtx* ctx, FUNCTOR functor, int64_t n, int64_t outer_size,
                                     int64_t bias_size, int64_t inner_size, const T* x,
                                     const T* bias, T* y) {
  if (IsKernelSafeInt32(n)) {
    FusedBiasAddForwardImpl<FUNCTOR, T, int32_t>(ctx, functor, outer_size, bias_size, inner_size, x,
                                                 bias, y);
  } else {
    FusedBiasAddForwardImpl<FUNCTOR, T, int64_t>(ctx, functor, outer_size, bias_size, inner_size, x,
                                                 bias, y);
  }
}

}  // namespace

template<typename T>
class FusedFusedBiasAddKernel final : public user_op::OpKernel {
 public:
  FusedFusedBiasAddKernel() = default;
  ~FusedFusedBiasAddKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* a_tensor = ctx->Tensor4ArgNameAndIndex("a", 0);
    const auto* b_tensor = ctx->Tensor4ArgNameAndIndex("b", 0);
    auto* out_tensor = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int32_t bias_add_axis = ctx->Attr<int32_t>("axis");
    const int64_t outer_size = a_tensor->shape().Count(0, bias_add_axis);
    const int64_t bias_size = a_tensor->shape().At(bias_add_axis);
    const int64_t inner_size = a_tensor->shape().Count(bias_add_axis + 1);
    const auto n = a_tensor->shape().elem_cnt();
    GeluFunctor<T> gelu_functor;
    DispatchFusedBiasAddForwardImpl<decltype(gelu_functor), T>(
        ctx->device_ctx(), gelu_functor, n, outer_size, bias_size, inner_size, a_tensor->dptr<T>(),
        b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(dtype)        \
  REGISTER_USER_KERNEL("fused_bias_add_gelu")             \
      .SetCreateFn<FusedFusedBiasAddKernel<dtype>>()      \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu") \
                       & (user_op::HobDataType("out", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(float)
REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(double)
REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(half)

template<typename T>
class FusedBiasAddMaskScaleKernel final : public user_op::OpKernel {
 public:
  FusedBiasAddMaskScaleKernel() = default;
  ~FusedBiasAddMaskScaleKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* a_tensor = ctx->Tensor4ArgNameAndIndex("a", 0);
    const auto* b_tensor = ctx->Tensor4ArgNameAndIndex("b", 0);
    const auto* mask_tensor = ctx->Tensor4ArgNameAndIndex("mask", 0);
    auto* out_tensor = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int32_t bias_add_axis = ctx->Attr<int32_t>("axis");
    const float scale = ctx->Attr<float>("scale");
    const int64_t outer_size = a_tensor->shape().Count(0, bias_add_axis);
    const int64_t bias_size = a_tensor->shape().At(bias_add_axis);
    const int64_t inner_size = a_tensor->shape().Count(bias_add_axis + 1);
    const auto n = a_tensor->shape().elem_cnt();
    if (ctx->user_op_conf().has_input("_add_to_output", 0)) {
      const user_op::Tensor* addend = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      MaskAndScaleAddFunctor<T> mask_and_scale_add_functor(mask_tensor->dptr<int8_t>(),
                                                           addend->dptr<T>(), scale);
      DispatchFusedBiasAddForwardImpl<decltype(mask_and_scale_add_functor), T>(
          ctx->device_ctx(), mask_and_scale_add_functor, n, outer_size, bias_size, inner_size,
          a_tensor->dptr<T>(), b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
    } else {
      MaskAndScaleFunctor<T> mask_and_scale_functor(mask_tensor->dptr<int8_t>(), scale);
      DispatchFusedBiasAddForwardImpl<decltype(mask_and_scale_functor), T>(
          ctx->device_ctx(), mask_and_scale_functor, n, outer_size, bias_size, inner_size,
          a_tensor->dptr<T>(), b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
    }
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(dtype)  \
  REGISTER_USER_KERNEL("fused_bias_add_mask_scale")       \
      .SetCreateFn<FusedBiasAddMaskScaleKernel<dtype>>()  \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu") \
                       & (user_op::HobDataType("out", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(float)
REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(double)
REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(half)

template<typename T>
class FusedFusedBiasAddGradKernel final : public user_op::OpKernel {
 public:
  FusedFusedBiasAddGradKernel() = default;
  ~FusedFusedBiasAddGradKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* a_tensor = ctx->Tensor4ArgNameAndIndex("a", 0);
    const auto* b_tensor = ctx->Tensor4ArgNameAndIndex("b", 0);
    const auto* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    auto* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const int32_t bias_add_axis = ctx->Attr<int32_t>("axis");
    const int64_t outer_size = a_tensor->shape().Count(0, bias_add_axis);
    const int64_t bias_size = a_tensor->shape().At(bias_add_axis);
    const int64_t inner_size = a_tensor->shape().Count(bias_add_axis + 1);
    const auto n = a_tensor->shape().elem_cnt();
    GeluGradFunctor<T> gelu_grad_functor;
    if (IsKernelSafeInt32(n)) {
      FusedBiasAddGradImpl<decltype(gelu_grad_functor), T, int32_t>(
          ctx->device_ctx(), gelu_grad_functor, outer_size, bias_size, inner_size,
          a_tensor->dptr<T>(), b_tensor->dptr<T>(), dy_tensor->dptr<T>(), dx_tensor->mut_dptr<T>());
    } else {
      FusedBiasAddGradImpl<decltype(gelu_grad_functor), T, int64_t>(
          ctx->device_ctx(), gelu_grad_functor, outer_size, bias_size, inner_size,
          a_tensor->dptr<T>(), b_tensor->dptr<T>(), dy_tensor->dptr<T>(), dx_tensor->mut_dptr<T>());
    }
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(dtype)   \
  REGISTER_USER_KERNEL("fused_bias_add_gelu_grad")        \
      .SetCreateFn<FusedFusedBiasAddGradKernel<dtype>>()  \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu") \
                       & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(float)
REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(double)
REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(half)

}  // namespace oneflow
