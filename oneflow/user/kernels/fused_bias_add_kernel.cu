#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#if CUDA_VERSION >= 11000
#include <hip/hip_bf16.h>
#endif  // CUDA_VERSION >= 11000
#include "oneflow/core/device/cuda_pseudo_bfloat16.h"

namespace oneflow {

namespace {

template<typename T>
struct GeluFunctor {
  __device__ T Compute(T x, int64_t i) const {
    return static_cast<T>(0.5) * x * (static_cast<T>(1.0) + erf(static_cast<T>(M_SQRT1_2) * x));
  }
};

template<>
struct GeluFunctor<half> {
  GeluFunctor<float> float_functor;
  __device__ half Compute(half x, int64_t i) const {
    return __float2half(float_functor.Compute(__half2float(x), i));
  }
  __device__ half2 ComputeHalf2(half2 x, int64_t i) const {
    half2 y;
    y.x = __float2half(float_functor.Compute(__half2float(x.x), 2 * i));
    y.y = __float2half(float_functor.Compute(__half2float(x.y), 2 * i + 1));
    return y;
  }
};

#if CUDA_VERSION >= 11000
template<>
struct GeluFunctor<hip_bfloat16> {
  GeluFunctor<float> float_functor;
  __device__ hip_bfloat16 Compute(hip_bfloat16 x, int64_t i) const {
    return static_cast<hip_bfloat16>(float_functor.Compute(static_cast<float>(x), i));
  }
};
#endif

template<typename T>
struct MaskAndScaleFunctor {
  MaskAndScaleFunctor(const bool* mask, float scale) : mask(mask), scale(scale) {}
  __device__ T Compute(T x, int64_t i) const { return x * static_cast<T>(mask[i] * scale); }
  const bool* mask;
  float scale;
};

template<>
struct MaskAndScaleFunctor<half> {
  MaskAndScaleFunctor(const bool* mask, float scale) : mask(mask), scale(scale) {}
  __device__ half Compute(half x, int64_t i) const {
    return x * static_cast<half>(mask[i] * scale);
  }
  __device__ half2 ComputeHalf2(half2 x, int64_t i) const {
    const char2* mask_c2 = reinterpret_cast<const char2*>(mask);
    char2 mask_val = mask_c2[i];
    half2 one_or_zero_h2;
    half2 h2_scale = __float2half2_rn(scale);
    one_or_zero_h2.x = mask_val.x;
    one_or_zero_h2.y = mask_val.y;
    return __hmul2(__hmul2(x, one_or_zero_h2), h2_scale);
  }
  const bool* mask;
  float scale;
};

template<typename T>
struct MaskAndScaleAddFunctor {
  MaskAndScaleAddFunctor(const bool* mask, const T* addend, float scale)
      : mask(mask), addend(addend), scale(scale) {}
  __device__ T Compute(T x, int64_t i) const {
    return x * static_cast<T>(mask[i] * scale) + addend[i];
  }
  const bool* mask;
  const T* addend;
  float scale;
};

template<>
struct MaskAndScaleAddFunctor<half> {
  MaskAndScaleAddFunctor(const bool* mask, const half* addend, float scale)
      : mask(mask), addend(addend), scale(scale) {}
  __device__ half Compute(half x, int64_t i) const {
    return x * static_cast<half>(mask[i] * scale) + addend[i];
  }
  __device__ half2 ComputeHalf2(half2 x, int64_t i) const {
    const char2* mask_c2 = reinterpret_cast<const char2*>(mask);
    const half2* addend_h2 = reinterpret_cast<const half2*>(addend);
    char2 mask_val = mask_c2[i];
    half2 one_or_zero_h2;
    half2 h2_scale = __float2half2_rn(scale);
    one_or_zero_h2.x = mask_val.x;
    one_or_zero_h2.y = mask_val.y;
    return __hadd2(__hmul2(__hmul2(x, one_or_zero_h2), h2_scale), addend_h2[i]);
  }
  const bool* mask;
  const half* addend;
  float scale;
};

template<typename T>
struct GeluGradFunctor {
  const T coef = std::sqrt(static_cast<T>(2.0) / std::acos(static_cast<T>(-1.0)));
  __device__ T Compute(T x, T dy, int64_t i) const {
    return static_cast<T>(0.5)
           * (static_cast<T>(1.0) + erf(static_cast<T>(M_SQRT1_2) * x)
              + x * coef * exp(static_cast<T>(-0.5) * x * x))
           * dy;
  }
};

template<>
struct GeluGradFunctor<half> {
  GeluGradFunctor<float> float_functor;
  __device__ half Compute(half x, half dy, int64_t i) const {
    return __float2half(float_functor.Compute(__half2float(x), __half2float(dy), i));
  }
};

#if CUDA_VERSION >= 11000
template<>
struct GeluGradFunctor<hip_bfloat16> {
  GeluGradFunctor<float> float_functor;
  __device__ hip_bfloat16 Compute(hip_bfloat16 x, hip_bfloat16 dy, int64_t i) const {
    return static_cast<hip_bfloat16>(
        float_functor.Compute(static_cast<float>(x), static_cast<float>(dy), i));
  }
};
#endif

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGpu(FUNCTOR functor, const Index elem_cnt, const Index bias_size,
                                const Index inner_size, const T* x, const T* bias, T* y) {
  const Index block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[(i % block_size) / inner_size];
    y[i] = functor.Compute(x_i, i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGradGpu(FUNCTOR grad_functor, const Index elem_cnt,
                                    const Index bias_size, const Index inner_size, const T* x,
                                    const T* bias, const T* dy, T* dx) {
  const Index block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[(i % block_size) / inner_size];
    dx[i] = grad_functor.Compute(x_i, dy[i], i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddRowGpu(FUNCTOR functor, const Index elem_cnt, const Index bias_size,
                                   const T* x, const T* bias, T* y) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i % bias_size];
    y[i] = functor.Compute(x_i, i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGradRowGpu(FUNCTOR grad_functor, const Index elem_cnt,
                                       const Index bias_size, const T* x, const T* bias,
                                       const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i % bias_size];
    dx[i] = grad_functor.Compute(x_i, dy[i], i);
  }
}

template<typename FUNCTOR, typename Index>
__global__ void FusedBiasAddRowGpuHalf2(FUNCTOR functor, const Index elem_cnt,
                                        const Index bias_size, const half* x, const half* bias,
                                        half* y) {
  const Index h2_elem_cnt = elem_cnt / 2;
  const Index h2_bias_size = bias_size / 2;
  const auto* x_h2 = reinterpret_cast<const half2*>(x);
  const auto* bias_h2 = reinterpret_cast<const half2*>(bias);
  auto* y_h2 = reinterpret_cast<half2*>(y);
  CUDA_1D_KERNEL_LOOP_T(Index, i, h2_elem_cnt) {
    half2 x_i = __hadd2(x_h2[i], bias_h2[i % h2_bias_size]);
    y_h2[i] = functor.ComputeHalf2(x_i, i);
  }
}

template<typename FUNCTOR, typename Index>
__global__ void FusedBiasAddGradRowGpuHalf2(FUNCTOR grad_functor, const Index elem_cnt,
                                            const Index bias_size, const half* x, const half* bias,
                                            const half* dy, half* dx) {
  const Index h2_elem_cnt = elem_cnt / 2;
  const Index h2_bias_size = bias_size / 2;
  const auto* x_h2 = reinterpret_cast<const half2*>(x);
  const auto* bias_h2 = reinterpret_cast<const half2*>(bias);
  const auto* dy_h2 = reinterpret_cast<const half2*>(dy);
  auto* dx_h2 = reinterpret_cast<half2*>(dx);
  CUDA_1D_KERNEL_LOOP_T(Index, i, h2_elem_cnt) {
    half2 x_i = __hadd2(x_h2[i], bias_h2[i % h2_bias_size]);
    half2 dy_i = dy_h2[i];
    half2 dx_i;
    dx_i.x = grad_functor.Compute(x_i.x, dy_i.x, 2 * i);
    dx_i.y = grad_functor.Compute(x_i.y, dy_i.y, 2 * i + 1);
    dx_h2[i] = dx_i;
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddColGpu(FUNCTOR functor, const Index elem_cnt, const Index inner_size,
                                   const T* x, const T* bias, T* y) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i / inner_size];
    y[i] = functor.Compute(x_i, i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGradColGpu(FUNCTOR grad_functor, const Index elem_cnt,
                                       const Index inner_size, const T* x, const T* bias,
                                       const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i / inner_size];
    dx[i] = grad_functor.Compute(x_i, dy[i], i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
struct FusedBiasAddRow {
  static void Invoke(ep::Stream* stream, FUNCTOR functor, Index elem_cnt, Index bias_size,
                     const T* x, const T* bias, T* y) {
    FusedBiasAddRowGpu<FUNCTOR, T, Index>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
           stream->As<ep::CudaStream>()->hip_stream()>>>(functor, elem_cnt, bias_size, x, bias, y);
  }
};

template<typename FUNCTOR, typename Index>
struct FusedBiasAddRow<FUNCTOR, half, Index> {
  static void Invoke(ep::Stream* stream, FUNCTOR functor, Index elem_cnt, Index bias_size,
                     const half* x, const half* bias, half* y) {
    if (bias_size % 2 == 0) {
      FusedBiasAddRowGpuHalf2<FUNCTOR, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt / 2), kCudaThreadsNumPerBlock, 0,
             stream->As<ep::CudaStream>()->hip_stream()>>>(functor, elem_cnt, bias_size, x, bias,
                                                            y);
    } else {
      FusedBiasAddRowGpu<FUNCTOR, half, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
             stream->As<ep::CudaStream>()->hip_stream()>>>(functor, elem_cnt, bias_size, x, bias,
                                                            y);
    }
  }
};

template<typename FUNCTOR, typename T, typename Index>
void FusedBiasAddForwardImpl(ep::Stream* stream, FUNCTOR functor, Index outer_size, Index bias_size,
                             Index inner_size, const T* x, const T* bias, T* y) {
  const Index elem_cnt = outer_size * bias_size * inner_size;
  if (inner_size == 1) {
    FusedBiasAddRow<FUNCTOR, T, Index>::Invoke(stream, functor, elem_cnt, bias_size, x, bias, y);
  } else if (outer_size == 1) {
    FusedBiasAddColGpu<FUNCTOR, T, Index><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock,
                                            0, stream->As<ep::CudaStream>()->hip_stream()>>>(
        functor, elem_cnt, inner_size, x, bias, y);
  } else {
    FusedBiasAddGpu<FUNCTOR, T, Index><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                         stream->As<ep::CudaStream>()->hip_stream()>>>(
        functor, elem_cnt, bias_size, inner_size, x, bias, y);
  }
}

template<typename FUNCTOR, typename T, typename Index>
struct FusedBiasAddGradRow {
  static void Invoke(ep::Stream* stream, FUNCTOR grad_functor, Index elem_cnt, Index bias_size,
                     const T* x, const T* bias, const T* dy, T* dx) {
    FusedBiasAddGradRowGpu<FUNCTOR, T, Index>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
           stream->As<ep::CudaStream>()->hip_stream()>>>(grad_functor, elem_cnt, bias_size, x,
                                                          bias, dy, dx);
  }
};

template<typename FUNCTOR, typename Index>
struct FusedBiasAddGradRow<FUNCTOR, half, Index> {
  static void Invoke(ep::Stream* stream, FUNCTOR grad_functor, Index elem_cnt, Index bias_size,
                     const half* x, const half* bias, const half* dy, half* dx) {
    if (bias_size % 2 == 0) {
      FusedBiasAddGradRowGpuHalf2<FUNCTOR, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt / 2), kCudaThreadsNumPerBlock, 0,
             stream->As<ep::CudaStream>()->hip_stream()>>>(grad_functor, elem_cnt, bias_size, x,
                                                            bias, dy, dx);
    } else {
      FusedBiasAddGradRowGpu<FUNCTOR, half, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
             stream->As<ep::CudaStream>()->hip_stream()>>>(grad_functor, elem_cnt, bias_size, x,
                                                            bias, dy, dx);
    }
  }
};

template<typename FUNCTOR, typename T, typename Index>
void FusedBiasAddGradImpl(ep::Stream* stream, FUNCTOR grad_functor, Index outer_size,
                          Index bias_size, Index inner_size, const T* x, const T* bias, const T* dy,
                          T* dx) {
  const Index elem_cnt = outer_size * bias_size * inner_size;
  if (inner_size == 1) {
    FusedBiasAddGradRow<FUNCTOR, T, Index>::Invoke(stream, grad_functor, elem_cnt, bias_size, x,
                                                   bias, dy, dx);
  } else if (outer_size == 1) {
    FusedBiasAddGradColGpu<FUNCTOR, T, Index>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
           stream->As<ep::CudaStream>()->hip_stream()>>>(grad_functor, elem_cnt, inner_size, x,
                                                          bias, dy, dx);
  } else {
    FusedBiasAddGradGpu<FUNCTOR, T, Index>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
           stream->As<ep::CudaStream>()->hip_stream()>>>(grad_functor, elem_cnt, bias_size,
                                                          inner_size, x, bias, dy, dx);
  }
}

template<typename FUNCTOR, typename T>
void DispatchFusedBiasAddForwardImpl(ep::Stream* stream, FUNCTOR functor, int64_t n,
                                     int64_t outer_size, int64_t bias_size, int64_t inner_size,
                                     const T* x, const T* bias, T* y) {
  if (IsKernelSafeInt32(n)) {
    FusedBiasAddForwardImpl<FUNCTOR, T, int32_t>(stream, functor, outer_size, bias_size, inner_size,
                                                 x, bias, y);
  } else {
    FusedBiasAddForwardImpl<FUNCTOR, T, int64_t>(stream, functor, outer_size, bias_size, inner_size,
                                                 x, bias, y);
  }
}

}  // namespace

template<typename T>
class FusedFusedBiasAddKernel final : public user_op::OpKernel {
 public:
  FusedFusedBiasAddKernel() = default;
  ~FusedFusedBiasAddKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* a_tensor = ctx->Tensor4ArgNameAndIndex("a", 0);
    const auto* b_tensor = ctx->Tensor4ArgNameAndIndex("b", 0);
    auto* out_tensor = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int32_t bias_add_axis = ctx->Attr<int32_t>("axis");
    const int64_t outer_size = a_tensor->shape_view().Count(0, bias_add_axis);
    const int64_t bias_size = a_tensor->shape_view().At(bias_add_axis);
    const int64_t inner_size = a_tensor->shape_view().Count(bias_add_axis + 1);
    const auto n = a_tensor->shape_view().elem_cnt();
    GeluFunctor<T> gelu_functor{};
    DispatchFusedBiasAddForwardImpl<decltype(gelu_functor), T>(
        ctx->stream(), gelu_functor, n, outer_size, bias_size, inner_size, a_tensor->dptr<T>(),
        b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(dtype)                     \
  REGISTER_USER_KERNEL("fused_bias_add_gelu")                          \
      .SetCreateFn<FusedFusedBiasAddKernel<dtype>>()                   \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(float)
REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(double)
REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(half)
#if CUDA_VERSION >= 11000
REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(hip_bfloat16)
#endif

template<typename T>
class FusedBiasAddMaskScaleKernel final : public user_op::OpKernel {
 public:
  FusedBiasAddMaskScaleKernel() = default;
  ~FusedBiasAddMaskScaleKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* a_tensor = ctx->Tensor4ArgNameAndIndex("a", 0);
    const auto* b_tensor = ctx->Tensor4ArgNameAndIndex("b", 0);
    const auto* mask_tensor = ctx->Tensor4ArgNameAndIndex("mask", 0);
    auto* out_tensor = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int32_t bias_add_axis = ctx->Attr<int32_t>("axis");
    const float scale = ctx->Attr<float>("scale");
    const int64_t outer_size = a_tensor->shape_view().Count(0, bias_add_axis);
    const int64_t bias_size = a_tensor->shape_view().At(bias_add_axis);
    const int64_t inner_size = a_tensor->shape_view().Count(bias_add_axis + 1);
    const auto n = a_tensor->shape_view().elem_cnt();
    if (ctx->has_input("_add_to_output", 0)) {
      const user_op::Tensor* addend = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      MaskAndScaleAddFunctor<T> mask_and_scale_add_functor(mask_tensor->dptr<bool>(),
                                                           addend->dptr<T>(), scale);
      DispatchFusedBiasAddForwardImpl<decltype(mask_and_scale_add_functor), T>(
          ctx->stream(), mask_and_scale_add_functor, n, outer_size, bias_size, inner_size,
          a_tensor->dptr<T>(), b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
    } else {
      MaskAndScaleFunctor<T> mask_and_scale_functor(mask_tensor->dptr<bool>(), scale);
      DispatchFusedBiasAddForwardImpl<decltype(mask_and_scale_functor), T>(
          ctx->stream(), mask_and_scale_functor, n, outer_size, bias_size, inner_size,
          a_tensor->dptr<T>(), b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
    }
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(dtype)               \
  REGISTER_USER_KERNEL("fused_bias_add_mask_scale")                    \
      .SetCreateFn<FusedBiasAddMaskScaleKernel<dtype>>()               \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(float)
REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(double)
REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(half)
#if CUDA_VERSION >= 11000
REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(hip_bfloat16)
#endif

template<typename T>
class FusedFusedBiasAddGradKernel final : public user_op::OpKernel {
 public:
  FusedFusedBiasAddGradKernel() = default;
  ~FusedFusedBiasAddGradKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* a_tensor = ctx->Tensor4ArgNameAndIndex("a", 0);
    const auto* b_tensor = ctx->Tensor4ArgNameAndIndex("b", 0);
    const auto* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    auto* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const int32_t bias_add_axis = ctx->Attr<int32_t>("axis");
    const int64_t outer_size = a_tensor->shape_view().Count(0, bias_add_axis);
    const int64_t bias_size = a_tensor->shape_view().At(bias_add_axis);
    const int64_t inner_size = a_tensor->shape_view().Count(bias_add_axis + 1);
    const auto n = a_tensor->shape_view().elem_cnt();
    GeluGradFunctor<T> gelu_grad_functor;
    if (IsKernelSafeInt32(n)) {
      FusedBiasAddGradImpl<decltype(gelu_grad_functor), T, int32_t>(
          ctx->stream(), gelu_grad_functor, outer_size, bias_size, inner_size, a_tensor->dptr<T>(),
          b_tensor->dptr<T>(), dy_tensor->dptr<T>(), dx_tensor->mut_dptr<T>());
    } else {
      FusedBiasAddGradImpl<decltype(gelu_grad_functor), T, int64_t>(
          ctx->stream(), gelu_grad_functor, outer_size, bias_size, inner_size, a_tensor->dptr<T>(),
          b_tensor->dptr<T>(), dy_tensor->dptr<T>(), dx_tensor->mut_dptr<T>());
    }
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(dtype)                \
  REGISTER_USER_KERNEL("fused_bias_add_gelu_grad")                     \
      .SetCreateFn<FusedFusedBiasAddGradKernel<dtype>>()               \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(float)
REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(double)
REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(half)
#if CUDA_VERSION >= 11000
REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(hip_bfloat16)
#endif

}  // namespace oneflow
