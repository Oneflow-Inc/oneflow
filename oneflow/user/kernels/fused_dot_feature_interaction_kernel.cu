#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/ep/include/primitive/copy_nd.h"
#include "oneflow/core/ep/include/primitive/batch_matmul.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include <mma.h>

namespace oneflow {

namespace {

__global__ void GenerateGatherIndicesGpu(const int32_t elem_cnt, const int32_t stride,
                                         const int32_t in_cols, const int32_t offset,
                                         int32_t* gather_indices) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const int32_t row = i / stride;
    const int32_t col = i - row * stride;
    if (col < row + offset) {
      int32_t in_index = row * in_cols + col;
      int32_t idx = row * (offset + row - 1 + offset) / 2 + col;
      gather_indices[idx] = in_index;
    }
  }
}

template<typename T>
__global__ void GatherConcatGpu(int32_t elem_cnt, int32_t out_cols, int32_t valid_out_cols,
                                int32_t in_cols, int32_t output_concat_end_dim,
                                const int32_t* gather_indices, const T* in,
                                const T* output_concat_ptr, T* out_ptr) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const int32_t row = i / out_cols;
    const int32_t col = i - row * out_cols;
    T out_val;
    if (col < output_concat_end_dim) {
      const int32_t output_concat_idx = row * output_concat_end_dim + col;
      out_val = output_concat_ptr[output_concat_idx];
    } else if (col < valid_out_cols) {
      const int32_t gather_col_idx = gather_indices[col - output_concat_end_dim];
      const int32_t in_offset = row * in_cols + gather_col_idx;
      out_val = in[in_offset];
    } else {
      out_val = 0;
    }
    out_ptr[i] = out_val;
  }
}

template<typename T>
__global__ void ScatterSplitAddTransposeGpu(int32_t elem_cnt, int32_t stride_dim, int32_t out_dim,
                                            int32_t in_grad_stride, int32_t in_grad_matrix_dim,
                                            int32_t in_grad_matrix_valid_dim,
                                            int32_t output_concat_end_dim, const int32_t offset,
                                            const T* dy, T* output_concat_grad, T* in_grad) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const int32_t row = i / stride_dim;
    const int32_t col = i - row * stride_dim;
    if (col < output_concat_end_dim) {
      output_concat_grad[row * output_concat_end_dim + col] = dy[row * out_dim + col];
    } else {
      int32_t in_col_id = col - output_concat_end_dim;
      const int32_t matrix_row = in_col_id / in_grad_matrix_dim;
      const int32_t matrix_col = in_col_id - matrix_row * in_grad_matrix_dim;
      T grad_val = 0;
      const T* row_dy = dy + row * out_dim + output_concat_end_dim;
      if (matrix_row < in_grad_matrix_valid_dim && matrix_col < in_grad_matrix_valid_dim) {
        if (matrix_col < matrix_row) {
          int32_t dy_col_idx = matrix_row * (offset + matrix_row - 1 + offset) / 2 + matrix_col;
          grad_val = row_dy[dy_col_idx];
        } else if (matrix_row < matrix_col) {
          // transpose add
          int32_t trans_row_id = matrix_col;
          int32_t trans_col_id = matrix_row;
          int32_t dy_col_idx =
              trans_row_id * (offset + trans_row_id - 1 + offset) / 2 + trans_col_id;
          grad_val = row_dy[dy_col_idx];
        } else if ((matrix_row == matrix_col) && (offset == 1)) {
          int32_t dy_col_idx = matrix_row * (offset + matrix_row - 1 + offset) / 2 + matrix_col;
          grad_val = row_dy[dy_col_idx] * static_cast<T>(2);
        }
      }
      int32_t in_grad_offset = row * in_grad_stride + in_col_id;
      in_grad[in_grad_offset] = grad_val;
    }
  }
}

template<typename T>
void ConcatFeatures(user_op::KernelComputeContext* ctx, int64_t dst_rows, int64_t dst_cols,
                    void* dst_ptr) {
  const int64_t feature_input_size = ctx->input_size("features");
  auto primitive = ep::primitive::NewPrimitive<ep::primitive::CopyNdFactory>(DeviceType::kCUDA, 2);
  DimVector dst_shape = {dst_rows, dst_cols};
  int64_t out_col_offset = 0;
  for (int64_t i = 0; i < feature_input_size; ++i) {
    const user_op::Tensor* feature = ctx->Tensor4ArgNameAndIndex("features", i);
    const int64_t feature_rows = feature->shape().At(0);
    const int64_t feature_cols = feature->shape().Count(1);
    DimVector dst_pos_vec = {0, out_col_offset};
    DimVector src_shape = {feature_rows, feature_cols};
    DimVector src_pos_vec = {0, 0};
    DimVector extent_vec = {feature_rows, feature_cols};
    primitive->Launch(ctx->stream(), feature->data_type(), 2, dst_ptr, dst_shape.data(),
                      dst_pos_vec.data(), feature->dptr<T>(), src_shape.data(), src_pos_vec.data(),
                      extent_vec.data());
    out_col_offset += feature_cols;
  }
  int64_t pad_dim = dst_cols - out_col_offset;
  if (pad_dim > 0) {
    char* out_ptr = reinterpret_cast<char*>(dst_ptr) + out_col_offset * sizeof(T);
    OF_CUDA_CHECK(hipMemset2DAsync(out_ptr, dst_cols * sizeof(T), 0, pad_dim * sizeof(T), dst_rows,
                                    ctx->stream()->As<ep::CudaStream>()->hip_stream()));
  }
}

template<typename T>
void GatherConcatKernel(ep::Stream* stream, int32_t elem_cnt, int32_t out_dim,
                        int32_t valid_out_dim, int32_t features_concated_dim,
                        int32_t concated_padded_dim, int32_t output_concat_end_dim,
                        bool self_interaction, const T* matmul_out, const T* output_concat_ptr,
                        int32_t* gather_indices_ptr, T* out_ptr) {
  hipStream_t hip_stream = stream->As<ep::CudaStream>()->hip_stream();
  const int32_t gen_indices_elem_cnt = features_concated_dim * features_concated_dim;
  int32_t offset = self_interaction ? 1 : 0;
  GenerateGatherIndicesGpu<<<BlocksNum4ThreadsNum(gen_indices_elem_cnt), kCudaThreadsNumPerBlock, 0,
                             hip_stream>>>(gen_indices_elem_cnt, features_concated_dim,
                                            concated_padded_dim, offset, gather_indices_ptr);

  int32_t matmul_stride = concated_padded_dim * concated_padded_dim;
  GatherConcatGpu<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, hip_stream>>>(
      elem_cnt, out_dim, valid_out_dim, matmul_stride, output_concat_end_dim, gather_indices_ptr,
      matmul_out, output_concat_ptr, out_ptr);
}

template<typename T>
void ScatterSplitAddTranspose(ep::Stream* stream, int32_t batch_size, int32_t out_dim,
                              int32_t concated_padded_dim, int32_t features_concated_dim,
                              int32_t output_concat_end_dim, const bool self_interaction,
                              const T* dy, T* output_concat_grad, T* matmul_out_grad_ptr) {
  int32_t stride_dim = output_concat_end_dim + concated_padded_dim * concated_padded_dim;
  int32_t matmul_stride = concated_padded_dim * concated_padded_dim;
  const int32_t elem_cnt = batch_size * stride_dim;
  int32_t offset = self_interaction ? 1 : 0;
  ScatterSplitAddTransposeGpu<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                stream->As<ep::CudaStream>()->hip_stream()>>>(
      elem_cnt, stride_dim, out_dim, matmul_stride, concated_padded_dim, features_concated_dim,
      output_concat_end_dim, offset, dy, output_concat_grad, matmul_out_grad_ptr);
}

template<typename T>
void ConcatFeaturesGrad(user_op::KernelComputeContext* ctx, const int64_t batch_size,
                        const int64_t concated_padded_dim, const int64_t vector_size,
                        const T* concated_features_grad) {
  auto primitive = ep::primitive::NewPrimitive<ep::primitive::CopyNdFactory>(DeviceType::kCUDA, 2);
  DimVector src_shape = {batch_size, concated_padded_dim * vector_size};
  int64_t in_col_offset = 0;
  for (int64_t i = 0; i < ctx->output_size("features_grad"); ++i) {
    user_op::Tensor* feature_grad = ctx->Tensor4ArgNameAndIndex("features_grad", i);
    const int64_t feature_grad_rows = feature_grad->shape().At(0);
    const int64_t feature_grad_cols = feature_grad->shape().Count(1);
    DimVector dst_shape = {feature_grad_rows, feature_grad_cols};
    DimVector dst_pos_vec = {0, 0};
    DimVector src_pos_vec = {0, in_col_offset};
    DimVector extent_vec = {feature_grad_rows, feature_grad_cols};
    in_col_offset += feature_grad_cols;
    primitive->Launch(ctx->stream(), feature_grad->data_type(), 2, feature_grad->mut_dptr(),
                      dst_shape.data(), dst_pos_vec.data(), concated_features_grad,
                      src_shape.data(), src_pos_vec.data(), extent_vec.data());
  }
}

template<typename T>
struct DefaultComputeType {
  using type = T;
};

template<>
struct DefaultComputeType<half> {
  using type = float;
};

template<typename T, size_t pack_size>
struct alignas(sizeof(T) * pack_size) Pack {
  T elem[pack_size];
};

int64_t GetPaddedDim(int64_t dim) {
  const int64_t align_dim = 16;
  const int64_t padded_dim =
      std::ceil(static_cast<float>(dim) / static_cast<float>(align_dim)) * align_dim;
  return padded_dim;
}

template<typename T, int32_t N>
struct DotFwdParam {
  const T* in[N];
  int32_t in_feature_dim[N];
  int32_t dim_start_offset[N];
  int32_t features_dim;
  const T* output_concat;
  int32_t output_concat_size;
  T* out;
  int32_t num_in;
};

constexpr int unroll_dim = 2;
template<typename T, typename ComputeType, int32_t N, int32_t pack_size, int TILE_DIM>
__global__ void DotFeatureInteractionTensorCore(
    int M_BLOCKS, int K_BLOCKS, int64_t batch_size, int padded_num_rows, int vector_num_pack,
    int padded_vector_num_pack, int out_num_cols, int out_num_cols_num_pack, int in_shared_mem_cols,
    int in_shared_mem_cols_num_pack, int acc_shared_mem_cols, int acc_shared_mem_cols_num_pack,
    int offset, int output_padding, DotFwdParam<T, N> param) {
  extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
  int warp_id = threadIdx.y;
  T* buf = reinterpret_cast<T*>(shared_buf);
  Pack<T, pack_size>* buf_pack = reinterpret_cast<Pack<T, pack_size>*>(shared_buf);
  ComputeType* acc_buf =
      reinterpret_cast<ComputeType*>(shared_buf + padded_num_rows * in_shared_mem_cols * sizeof(T));
  int batch_idx = blockIdx.x;
  T* batch_out = param.out + batch_idx * out_num_cols;
  Pack<T, pack_size>* batch_out_pack =
      reinterpret_cast<Pack<T, pack_size>*>(param.out) + batch_idx * out_num_cols_num_pack;
  const int output_concat_size = param.output_concat_size;
  const T* batch_output_concat =
      (param.output_concat) ? (param.output_concat + batch_idx * output_concat_size) : nullptr;
  for (int col = threadIdx.x; col < vector_num_pack; col += blockDim.x) {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      if (i >= param.num_in) { break; }
      const Pack<T, pack_size>* batch_in = reinterpret_cast<const Pack<T, pack_size>*>(param.in[i])
                                           + batch_idx * param.in_feature_dim[i] * vector_num_pack;
      for (int j = threadIdx.y * unroll_dim; j < param.in_feature_dim[i];
           j += blockDim.y * unroll_dim) {
#pragma unroll
        for (int k = 0; k < unroll_dim; ++k) {
          int in_row = j + k;
          if (in_row >= param.in_feature_dim[i]) { break; }
          int buf_row = param.dim_start_offset[i] + in_row;
          buf_pack[buf_row * in_shared_mem_cols_num_pack + col] =
              batch_in[in_row * vector_num_pack + col];
        }
      }
    }
  }
  Pack<T, pack_size> zero;
  for (int k = 0; k < pack_size; ++k) { zero.elem[k] = 0; }
  for (int row = threadIdx.y; row < param.features_dim; row += blockDim.y) {
    for (int col = vector_num_pack + threadIdx.x; col < padded_vector_num_pack; col += blockDim.x) {
      buf_pack[row * in_shared_mem_cols_num_pack + col] = zero;
    }
  }
  __syncthreads();
  for (int blocks_id = warp_id; blocks_id < M_BLOCKS * M_BLOCKS; blocks_id += blockDim.y) {
    int blocks_row_id = blocks_id / M_BLOCKS;
    int blocks_col_id = blocks_id - blocks_row_id * M_BLOCKS;
    if (blocks_row_id >= blocks_col_id) {
      nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TILE_DIM, TILE_DIM, TILE_DIM, ComputeType>
          acc;
      nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TILE_DIM, TILE_DIM, TILE_DIM, T,
                             nvcuda::wmma::row_major>
          a;
      nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TILE_DIM, TILE_DIM, TILE_DIM, T,
                             nvcuda::wmma::col_major>
          b;
      nvcuda::wmma::fill_fragment(acc, 0.0f);
      for (int step = 0; step < K_BLOCKS; ++step) {
        T* tile_a_ptr = buf + blocks_row_id * TILE_DIM * in_shared_mem_cols + step * TILE_DIM;
        T* tile_b_ptr = buf + blocks_col_id * TILE_DIM * in_shared_mem_cols + step * TILE_DIM;
        nvcuda::wmma::load_matrix_sync(a, tile_a_ptr, in_shared_mem_cols);
        nvcuda::wmma::load_matrix_sync(b, tile_b_ptr, in_shared_mem_cols);
        nvcuda::wmma::mma_sync(acc, a, b, acc);
      }
      ComputeType* tile_ptr =
          acc_buf + blocks_row_id * TILE_DIM * acc_shared_mem_cols + blocks_col_id * TILE_DIM;
      nvcuda::wmma::store_matrix_sync(tile_ptr, acc, acc_shared_mem_cols,
                                      nvcuda::wmma::mem_row_major);
    }
  }
  __syncthreads();
  T* emb_out = batch_out + output_concat_size;
  for (int base_row = threadIdx.y * unroll_dim; base_row < param.features_dim;
       base_row += unroll_dim * blockDim.y) {
#pragma unroll
    for (int k = 0; k < unroll_dim; ++k) {
      int row = base_row + k;
      if (row >= param.features_dim) { break; }
      for (int col = threadIdx.x; col < param.features_dim; col += blockDim.x) {
        if (col < row + offset) {
          int64_t idx = row * (offset + row - 1 + offset) / 2 + col;
          emb_out[idx] = static_cast<T>(acc_buf[row * acc_shared_mem_cols + col]);
        }
      }
    }
  }
  int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
  for (int i = thread_id; i < output_concat_size; i += blockDim.x * blockDim.y) {
    batch_out[i] = batch_output_concat[i];
  }
  for (int i = thread_id; i < output_padding; i += blockDim.x * blockDim.y) {
    batch_out[out_num_cols - 1 - i] = 0;
  }
}

template<typename T, int N, int32_t pack_size>
struct DotFeatureInteractionKernel {
  static bool Launch(ep::Stream* stream, int64_t batch_size, int concated_padded_dim,
                     int vector_size, int out_num_cols, bool self_interaction, int output_padding,
                     const DotFwdParam<T, N>& param) {
    UNIMPLEMENTED();
    return false;
  }
};

template<int N, int32_t pack_size>
struct DotFeatureInteractionKernel<half, N, pack_size> {
  static bool Launch(ep::Stream* stream, int64_t batch_size, int concated_padded_dim,
                     int vector_size, int out_num_cols, bool self_interaction, int output_padding,
                     const DotFwdParam<half, N>& param) {
    const int block_size = 128;
    const int block_dim_x = 32;
    const int block_dim_y = block_size / block_dim_x;
    const int num_blocks = batch_size;
    const int TILE_DIM = 16;
    const int64_t padded_vector_size = GetPaddedDim(vector_size);
    const int M_BLOCKS = concated_padded_dim / TILE_DIM;
    const int K_BLOCKS = padded_vector_size / TILE_DIM;
    const int skew_half = 8;
    const int skew_acc = 8;  // consider adjust this
    const int in_shared_mem_num_cols = padded_vector_size + skew_half;
    const int acc_shared_mem_num_cols = concated_padded_dim + skew_acc;
    const size_t in_shared_mem_bytes = concated_padded_dim * in_shared_mem_num_cols * sizeof(half);
    using ComputeType = typename DefaultComputeType<half>::type;
    const size_t acc_shared_mem_bytes =
        concated_padded_dim * acc_shared_mem_num_cols * sizeof(ComputeType);
    const size_t total_shared_mem_bytes = in_shared_mem_bytes + acc_shared_mem_bytes;
    const int32_t offset = self_interaction ? 1 : 0;
    const int out_num_cols_num_pack = out_num_cols / pack_size;
    const int vector_num_pack = vector_size / pack_size;
    const int padded_vector_num_pack = padded_vector_size / pack_size;
    const int in_shared_mem_cols_num_pack = in_shared_mem_num_cols / pack_size;
    const int acc_shared_mem_cols_num_pack = acc_shared_mem_num_cols / pack_size;
    int max_active_blocks;
    OF_CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks,
        DotFeatureInteractionTensorCore<half, ComputeType, N, pack_size, TILE_DIM>, block_size,
        total_shared_mem_bytes));
    if (max_active_blocks <= 0) { return false; }
    hipStream_t hip_stream = stream->As<ep::CudaStream>()->hip_stream();
    DotFeatureInteractionTensorCore<half, ComputeType, N, pack_size, TILE_DIM>
        <<<num_blocks, dim3(block_dim_x, block_dim_y), total_shared_mem_bytes, hip_stream>>>(
            M_BLOCKS, K_BLOCKS, batch_size, concated_padded_dim, vector_num_pack,
            padded_vector_num_pack, out_num_cols, out_num_cols_num_pack, in_shared_mem_num_cols,
            in_shared_mem_cols_num_pack, acc_shared_mem_num_cols, acc_shared_mem_cols_num_pack,
            offset, output_padding, param);
    return true;
  }
};

template<typename T, int32_t N>
struct DotBwdParam {
  const T* dy;
  const T* in[N];
  T* in_grad[N];
  T* output_concat_grad;
  int32_t output_concat_size;
  int32_t in_feature_dim[N];
  int32_t dim_start_offset[N];
  int32_t features_dim;
  int32_t num_in;
};

template<typename T, typename ComputeType, int32_t N, int32_t pack_size, int TILE_DIM>
__global__ void DotFeatureInteractionBackwardTensorCore(
    int M_BLOCKS, int N_BLOCKS, int K_BLOCKS, int64_t batch_size, int padded_num_rows,
    int vector_num_pack, int padded_vector_num_pack, int out_num_cols, int out_num_cols_num_pack,
    int in_shared_mem_cols, int in_shared_mem_cols_num_pack, int matrix_dy_shared_mem_cols,
    int matrix_dy_shared_mem_cols_num_pack, int offset, DotBwdParam<T, N> param) {
  extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
  int warp_id = threadIdx.y;
  T* in_buf = reinterpret_cast<T*>(shared_buf);
  Pack<T, pack_size>* in_buf_pack = reinterpret_cast<Pack<T, pack_size>*>(shared_buf);
  T* matrix_dy_buf = in_buf + padded_num_rows * in_shared_mem_cols;
  Pack<T, pack_size>* matrix_dy_pack = reinterpret_cast<Pack<T, pack_size>*>(matrix_dy_buf);
  ComputeType* in_grad_buf =
      reinterpret_cast<ComputeType*>(matrix_dy_buf + padded_num_rows * matrix_dy_shared_mem_cols);
  Pack<ComputeType, pack_size>* in_grad_buf_pack =
      reinterpret_cast<Pack<ComputeType, pack_size>*>(in_grad_buf);

  int batch_idx = blockIdx.x;
  const T* batch_dy = param.dy + batch_idx * out_num_cols;
  const Pack<T, pack_size>* batch_dy_pack =
      reinterpret_cast<const Pack<T, pack_size>*>(param.dy) + batch_idx * out_num_cols_num_pack;
  const int output_concat_size = param.output_concat_size;
  T* batch_output_concat_grad = (param.output_concat_grad)
                                    ? (param.output_concat_grad + batch_idx * output_concat_size)
                                    : nullptr;
  int features_dim = param.features_dim;
  // 1.split dy to concat_out_grad and matrix_dy buf
  int thread_id = threadIdx.x + threadIdx.y * blockDim.x;
  for (int i = thread_id; i < output_concat_size; i += blockDim.x * blockDim.y) {
    batch_output_concat_grad[i] = batch_dy[i];
  }
  const T* batch_interaction_dy = batch_dy + output_concat_size;
  for (int matrix_row = threadIdx.y; matrix_row < padded_num_rows; matrix_row += blockDim.y) {
    for (int matrix_col = threadIdx.x; matrix_col < padded_num_rows; matrix_col += blockDim.x) {
      const int64_t i = matrix_row * matrix_dy_shared_mem_cols + matrix_col;
      T grad_val = 0;
      if (matrix_row < features_dim && matrix_col < features_dim) {
        if (matrix_col < matrix_row) {
          int32_t dy_col_idx = matrix_row * (offset + matrix_row - 1 + offset) / 2 + matrix_col;
          grad_val = batch_interaction_dy[dy_col_idx];
        } else if (matrix_row < matrix_col) {
          // transpose add
          int32_t trans_row_id = matrix_col;
          int32_t trans_col_id = matrix_row;
          int32_t dy_col_idx =
              trans_row_id * (offset + trans_row_id - 1 + offset) / 2 + trans_col_id;
          grad_val = batch_interaction_dy[dy_col_idx];
        } else if ((matrix_row == matrix_col) && (offset == 1)) {
          int32_t dy_col_idx = matrix_row * (offset + matrix_row - 1 + offset) / 2 + matrix_col;
          grad_val = batch_interaction_dy[dy_col_idx] * static_cast<T>(2);
        }
      }
      matrix_dy_buf[i] = grad_val;
    }
  }

  // 2.load in to in in_buf
  for (int col = threadIdx.x; col < vector_num_pack; col += blockDim.x) {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      if (i >= param.num_in) { break; }
      const Pack<T, pack_size>* batch_in = reinterpret_cast<const Pack<T, pack_size>*>(param.in[i])
                                           + batch_idx * param.in_feature_dim[i] * vector_num_pack;
      for (int j = threadIdx.y * unroll_dim; j < param.in_feature_dim[i];
           j += blockDim.y * unroll_dim) {
#pragma unroll
        for (int k = 0; k < unroll_dim; ++k) {
          int in_row = j + k;
          if (in_row >= param.in_feature_dim[i]) { break; }
          int buf_row = param.dim_start_offset[i] + in_row;
          in_buf_pack[buf_row * in_shared_mem_cols_num_pack + col] =
              batch_in[in_row * vector_num_pack + col];
        }
      }
    }
  }
  Pack<T, pack_size> zero;
  for (int k = 0; k < pack_size; ++k) { zero.elem[k] = 0; }
#pragma unroll
  for (int row = features_dim + threadIdx.y; row < padded_num_rows; row += blockDim.y) {
    for (int col = threadIdx.x; col < padded_vector_num_pack; col += blockDim.x) {
      in_buf_pack[row * in_shared_mem_cols_num_pack + col] = zero;
    }
  }
  for (int row = threadIdx.y; row < features_dim; row += blockDim.y) {
    for (int col = vector_num_pack + threadIdx.x; col < padded_vector_num_pack; col += blockDim.x) {
      in_buf_pack[row * in_shared_mem_cols_num_pack + col] = zero;
    }
  }
  __syncthreads();

  for (int blocks_id = warp_id; blocks_id < M_BLOCKS * N_BLOCKS; blocks_id += blockDim.y) {
    int blocks_row_id = blocks_id / N_BLOCKS;
    int blocks_col_id = blocks_id - blocks_row_id * N_BLOCKS;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TILE_DIM, TILE_DIM, TILE_DIM, ComputeType>
        acc;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TILE_DIM, TILE_DIM, TILE_DIM, T,
                           nvcuda::wmma::row_major>
        a;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TILE_DIM, TILE_DIM, TILE_DIM, T,
                           nvcuda::wmma::row_major>
        b;
    nvcuda::wmma::fill_fragment(acc, 0.0f);
    for (int step = 0; step < K_BLOCKS; ++step) {
      // blocks_row_id is a row_id, step is a col_id. blocks_col_id is b col_id,
      // step is b row_id.
      T* tile_a_ptr =
          matrix_dy_buf + blocks_row_id * TILE_DIM * matrix_dy_shared_mem_cols + step * TILE_DIM;
      T* tile_b_ptr = in_buf + step * TILE_DIM * in_shared_mem_cols + blocks_col_id * TILE_DIM;
      nvcuda::wmma::load_matrix_sync(a, tile_a_ptr, matrix_dy_shared_mem_cols);
      nvcuda::wmma::load_matrix_sync(b, tile_b_ptr, in_shared_mem_cols);
      nvcuda::wmma::mma_sync(acc, a, b, acc);
    }
    ComputeType* tile_ptr =
        in_grad_buf + blocks_row_id * TILE_DIM * in_shared_mem_cols + blocks_col_id * TILE_DIM;
    nvcuda::wmma::store_matrix_sync(tile_ptr, acc, in_shared_mem_cols, nvcuda::wmma::mem_row_major);
  }
  __syncthreads();

  // 4.split in_grad buf to dx
  for (int col = threadIdx.x; col < vector_num_pack; col += blockDim.x) {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      if (i >= param.num_in) { break; }
      Pack<T, pack_size>* batch_in_grad = reinterpret_cast<Pack<T, pack_size>*>(param.in_grad[i])
                                          + batch_idx * param.in_feature_dim[i] * vector_num_pack;
      for (int j = threadIdx.y * unroll_dim; j < param.in_feature_dim[i];
           j += blockDim.y * unroll_dim) {
#pragma unroll
        for (int k = 0; k < unroll_dim; ++k) {
          int in_row = j + k;
          if (in_row >= param.in_feature_dim[i]) { break; }
          int buf_row = param.dim_start_offset[i] + in_row;
          Pack<T, pack_size> grad_val;
          Pack<ComputeType, pack_size> buf_grad_val =
              in_grad_buf_pack[buf_row * in_shared_mem_cols_num_pack + col];
          for (int t = 0; t < pack_size; ++t) {
            grad_val.elem[t] = static_cast<T>(buf_grad_val.elem[t]);
          }
          batch_in_grad[in_row * vector_num_pack + col] = grad_val;
        }
      }
    }
  }
}

template<typename T, int N, int32_t pack_size>
struct DotFeatureInteractionBackwardKernel {
  static bool Launch(ep::Stream* stream, int64_t batch_size, int concated_padded_dim,
                     int vector_size, int out_num_cols, bool self_interaction,
                     const DotBwdParam<T, N>& param) {
    UNIMPLEMENTED();
    return false;
  }
};

template<int N, int32_t pack_size>
struct DotFeatureInteractionBackwardKernel<half, N, pack_size> {
  static bool Launch(ep::Stream* stream, int64_t batch_size, int concated_padded_dim,
                     int vector_size, int out_num_cols, bool self_interaction,
                     const DotBwdParam<half, N>& param) {
    const int block_size = 256;
    const int block_dim_x = 32;
    const int block_dim_y = block_size / block_dim_x;
    const int num_blocks = batch_size;
    const int TILE_DIM = 16;
    const int64_t padded_vector_size = GetPaddedDim(vector_size);
    const int M_BLOCKS = concated_padded_dim / TILE_DIM;
    const int K_BLOCKS = concated_padded_dim / TILE_DIM;
    const int N_BLOCKS = padded_vector_size / TILE_DIM;
    const int skew_half = 8;
    const int in_shared_mem_num_cols = padded_vector_size + skew_half;
    const int matrix_dy_shared_mem_cols = concated_padded_dim + skew_half;
    const size_t in_shared_mem_bytes = concated_padded_dim * in_shared_mem_num_cols * sizeof(half);
    const size_t matrix_dy_shared_mem_bytes =
        concated_padded_dim * matrix_dy_shared_mem_cols * sizeof(half);
    using ComputeType = typename DefaultComputeType<half>::type;
    const size_t in_grad_shared_mem_bytes =
        concated_padded_dim * in_shared_mem_num_cols * sizeof(ComputeType);
    const size_t total_shared_mem_bytes =
        in_shared_mem_bytes + matrix_dy_shared_mem_bytes + in_grad_shared_mem_bytes;
    const int32_t offset = self_interaction ? 1 : 0;
    const int out_num_cols_num_pack = out_num_cols / pack_size;
    const int vector_num_pack = vector_size / pack_size;
    const int padded_vector_num_pack = padded_vector_size / pack_size;
    const int in_shared_mem_cols_num_pack = in_shared_mem_num_cols / pack_size;
    const int matrix_dy_shared_mem_cols_num_pack = matrix_dy_shared_mem_cols / pack_size;
    int max_active_blocks;
    OF_CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks,
        DotFeatureInteractionBackwardTensorCore<half, ComputeType, N, pack_size, TILE_DIM>,
        block_size, total_shared_mem_bytes));
    if (max_active_blocks <= 0) { return false; }
    hipStream_t hip_stream = stream->As<ep::CudaStream>()->hip_stream();
    DotFeatureInteractionBackwardTensorCore<half, ComputeType, N, pack_size, TILE_DIM>
        <<<num_blocks, dim3(block_dim_x, block_dim_y), total_shared_mem_bytes, hip_stream>>>(
            M_BLOCKS, N_BLOCKS, K_BLOCKS, batch_size, concated_padded_dim, vector_num_pack,
            padded_vector_num_pack, out_num_cols, out_num_cols_num_pack, in_shared_mem_num_cols,
            in_shared_mem_cols_num_pack, matrix_dy_shared_mem_cols,
            matrix_dy_shared_mem_cols_num_pack, offset, param);

    return true;
  }
};

template<typename T, int N>
bool DispatchFeatureInteractionDotPackSize(user_op::KernelComputeContext* ctx,
                                           const int32_t input_size) {
  CHECK_LE(input_size, N) << input_size;
  user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
  const int64_t batch_size = out->shape().At(0);
  const int64_t out_num_cols = out->shape().At(1);
  const int64_t vector_size = ctx->TensorDesc4ArgNameAndIndex("features", 0)->shape().At(2);
  DotFwdParam<T, N> param;
  param.num_in = input_size;
  param.out = out->mut_dptr<T>();
  int64_t features_concated_dim = 0;
  for (int i = 0; i < input_size; ++i) {
    param.in[i] = ctx->Tensor4ArgNameAndIndex("features", i)->dptr<T>();
    param.in_feature_dim[i] = ctx->TensorDesc4ArgNameAndIndex("features", i)->shape().At(1);
    param.dim_start_offset[i] = features_concated_dim;
    features_concated_dim += param.in_feature_dim[i];
  }
  const int64_t concated_padded_dim = GetPaddedDim(features_concated_dim);
  param.features_dim = features_concated_dim;
  if (ctx->has_input("output_concat", 0)) {
    const user_op::Tensor* output_concat = ctx->Tensor4ArgNameAndIndex("output_concat", 0);
    param.output_concat = output_concat->dptr<T>();
    param.output_concat_size = output_concat->shape().At(1);
  } else {
    param.output_concat = nullptr;
    param.output_concat_size = 0;
  }
  const bool self_interaction = ctx->Attr<bool>("self_interaction");
  const int32_t output_padding = ctx->Attr<int32_t>("output_padding");
  if (vector_size % 4 == 0 && out_num_cols % 4 == 0) {
    return DotFeatureInteractionKernel<T, N, 4>::Launch(
        ctx->stream(), batch_size, concated_padded_dim, vector_size, out_num_cols, self_interaction,
        output_padding, param);
  } else if (vector_size % 2 == 0 && out_num_cols % 2 == 0) {
    return DotFeatureInteractionKernel<T, N, 2>::Launch(
        ctx->stream(), batch_size, concated_padded_dim, vector_size, out_num_cols, self_interaction,
        output_padding, param);
  } else {
    return DotFeatureInteractionKernel<T, N, 1>::Launch(
        ctx->stream(), batch_size, concated_padded_dim, vector_size, out_num_cols, self_interaction,
        output_padding, param);
  }
}

template<typename T, int N>
bool DispatchFeatureInteractionDotBackwardPackSize(user_op::KernelComputeContext* ctx,
                                                   const int32_t input_size) {
  CHECK_LE(input_size, N) << input_size;
  user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
  const int64_t batch_size = dy->shape().At(0);
  const int64_t out_num_cols = dy->shape().At(1);
  const int64_t vector_size = ctx->TensorDesc4ArgNameAndIndex("features", 0)->shape().At(2);
  DotBwdParam<T, N> param;
  param.num_in = input_size;
  param.dy = dy->dptr<T>();
  int64_t features_concated_dim = 0;
  for (int i = 0; i < input_size; ++i) {
    param.in[i] = ctx->Tensor4ArgNameAndIndex("features", i)->dptr<T>();
    param.in_grad[i] = ctx->Tensor4ArgNameAndIndex("features_grad", i)->mut_dptr<T>();
    param.in_feature_dim[i] = ctx->TensorDesc4ArgNameAndIndex("features", i)->shape().At(1);
    param.dim_start_offset[i] = features_concated_dim;
    features_concated_dim += param.in_feature_dim[i];
  }
  const int64_t concated_padded_dim = GetPaddedDim(features_concated_dim);
  param.features_dim = features_concated_dim;
  if (ctx->has_output("output_concat_grad", 0)) {
    user_op::Tensor* output_concat_grad = ctx->Tensor4ArgNameAndIndex("output_concat_grad", 0);
    param.output_concat_grad = output_concat_grad->mut_dptr<T>();
    param.output_concat_size = output_concat_grad->shape().At(1);
  } else {
    param.output_concat_grad = nullptr;
    param.output_concat_size = 0;
  }
  const bool self_interaction = ctx->Attr<bool>("self_interaction");
  if (vector_size % 4 == 0 && out_num_cols % 4 == 0) {
    return DotFeatureInteractionBackwardKernel<T, N, 4>::Launch(
        ctx->stream(), batch_size, concated_padded_dim, vector_size, out_num_cols, self_interaction,
        param);
  } else if (vector_size % 2 == 0 && out_num_cols % 2 == 0) {
    return DotFeatureInteractionBackwardKernel<T, N, 2>::Launch(
        ctx->stream(), batch_size, concated_padded_dim, vector_size, out_num_cols, self_interaction,
        param);
  } else {
    return DotFeatureInteractionBackwardKernel<T, N, 1>::Launch(
        ctx->stream(), batch_size, concated_padded_dim, vector_size, out_num_cols, self_interaction,
        param);
  }
}

template<typename T, int32_t N>
struct Param {
  const T* in[N];
  int32_t in_feature_dim[N];
  T* out;
  int32_t num_in;
};

template<typename T, int32_t N, int32_t pack_size>
__global__ void FeatureInteractionSum(int64_t batch_size, int64_t vector_num_pack,
                                      Param<T, N> param) {
  using ComputeType = typename DefaultComputeType<T>::type;
  Pack<T, pack_size>* dst_pack = reinterpret_cast<Pack<T, pack_size>*>(param.out);
  for (int batch_idx = blockIdx.x * blockDim.y + threadIdx.y; batch_idx < batch_size;
       batch_idx += gridDim.x * blockDim.y) {
    Pack<T, pack_size>* batch_out = dst_pack + batch_idx * vector_num_pack;
    for (int col_id = threadIdx.x; col_id < vector_num_pack; col_id += blockDim.x) {
      Pack<ComputeType, pack_size> sum;
      Pack<ComputeType, pack_size> square_sum;
#pragma unroll
      for (int k = 0; k < pack_size; ++k) {
        sum.elem[k] = static_cast<ComputeType>(0);
        square_sum.elem[k] = static_cast<ComputeType>(0);
      }
      for (int i = 0; i < N; ++i) {
        if (i >= param.num_in) { break; }
        const Pack<T, pack_size>* batch_in =
            reinterpret_cast<const Pack<T, pack_size>*>(param.in[i])
            + batch_idx * param.in_feature_dim[i] * vector_num_pack;
#pragma unroll
        for (int j = 0; j < param.in_feature_dim[i]; ++j) {
          Pack<T, pack_size> val = batch_in[j * vector_num_pack + col_id];
#pragma unroll
          for (int k = 0; k < pack_size; ++k) {
            const ComputeType compute_val = static_cast<ComputeType>(val.elem[k]);
            sum.elem[k] += compute_val;
            square_sum.elem[k] += compute_val * compute_val;
          }
        }
      }
      Pack<T, pack_size> out;
#pragma unroll
      for (int k = 0; k < pack_size; ++k) {
        out.elem[k] = static_cast<T>((sum.elem[k] * sum.elem[k] - square_sum.elem[k])
                                     * static_cast<ComputeType>(0.5));
      }
      batch_out[col_id] = out;
    }
  }
}

template<typename T, int32_t N>
struct GradParam {
  const T* dy;
  const T* in[N];
  int32_t in_feature_dim[N];
  T* in_grad[N];
  int32_t num_in;
};

template<typename T, int32_t N>
__global__ void FeatureInteractionSumGrad(int64_t batch_size, int64_t vector_size,
                                          GradParam<T, N> param) {
  using ComputeType = typename DefaultComputeType<T>::type;
  for (int batch_idx = blockIdx.x * blockDim.y + threadIdx.y; batch_idx < batch_size;
       batch_idx += gridDim.x * blockDim.y) {
    const T* batch_dy = param.dy + batch_idx * vector_size;
    for (int col_id = threadIdx.x; col_id < vector_size; col_id += blockDim.x) {
      ComputeType sum = 0;
      for (int i = 0; i < N; ++i) {
        if (i >= param.num_in) { break; }
        const T* batch_in = param.in[i] + batch_idx * param.in_feature_dim[i] * vector_size;
        for (int j = 0; j < param.in_feature_dim[i]; ++j) {
          sum += static_cast<ComputeType>(batch_in[j * vector_size + col_id]);
        }
      }
      for (int i = 0; i < N; ++i) {
        if (i >= param.num_in) { break; }
        const int64_t in_batch_offset = batch_idx * param.in_feature_dim[i] * vector_size;
        const T* batch_in = param.in[i] + in_batch_offset;
        T* batch_in_grad = param.in_grad[i] + in_batch_offset;
        for (int j = 0; j < param.in_feature_dim[i]; ++j) {
          const int64_t offset = j * vector_size + col_id;
          batch_in_grad[offset] =
              static_cast<T>(static_cast<ComputeType>(batch_dy[col_id])
                             * (sum - static_cast<ComputeType>(batch_in[offset])));
        }
      }
    }
  }
}

void GetBlockDims(const int64_t vector_size, int* block_dim_x, int* block_dim_y) {
  const int block_size = 256;
  if (vector_size < block_size) {
    *block_dim_x = std::ceil(static_cast<float>(vector_size) / 8) * 8;
    *block_dim_y = (block_size + *block_dim_x - 1) / *block_dim_x;
  } else {
    *block_dim_x = block_size;
    *block_dim_y = 1;
  }
}

int GetNumBlocks(const int64_t num_instances, const int64_t instance_per_block) {
  int max_blocks = (num_instances + instance_per_block - 1) / instance_per_block;
  return std::min(max_blocks, kCudaMaxBlocksNum);
}

template<typename T, int32_t N>
void DispatchFeatureInteractionSumPackSize(ep::Stream* stream, const int64_t batch_size,
                                           const int64_t vector_size, const Param<T, N>& param) {
  int block_dim_x;
  int block_dim_y;
  const int pack_size = (vector_size % 2 == 0) ? 2 : 1;
  const int64_t vector_num_pack = vector_size / pack_size;
  GetBlockDims(vector_num_pack, &block_dim_x, &block_dim_y);
  const int num_blocks = GetNumBlocks(batch_size, block_dim_y);
  dim3 block_dims = dim3(block_dim_x, block_dim_y);
  hipStream_t hip_stream = stream->As<ep::CudaStream>()->hip_stream();
  if (pack_size == 2) {
    FeatureInteractionSum<T, N, 2>
        <<<num_blocks, block_dims, 0, hip_stream>>>(batch_size, vector_num_pack, param);
  } else {
    FeatureInteractionSum<T, N, 1>
        <<<num_blocks, block_dims, 0, hip_stream>>>(batch_size, vector_num_pack, param);
  }
}

template<typename T, int N>
void DispatchFeatureInteractionSumInputSize(user_op::KernelComputeContext* ctx,
                                            const int32_t input_size) {
  CHECK_LE(input_size, N) << input_size;
  user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
  const int64_t batch_size = out->shape().At(0);
  const int64_t vector_size = out->shape().At(1);
  Param<T, N> param;
  param.num_in = input_size;
  param.out = out->mut_dptr<T>();
  for (int i = 0; i < input_size; ++i) {
    param.in[i] = ctx->Tensor4ArgNameAndIndex("features", i)->dptr<T>();
    param.in_feature_dim[i] = ctx->TensorDesc4ArgNameAndIndex("features", i)->shape().At(1);
  }
  DispatchFeatureInteractionSumPackSize<T, N>(ctx->stream(), batch_size, vector_size, param);
}

template<typename T, int N>
void DispatchFeatureInteractionSumGradInputSize(user_op::KernelComputeContext* ctx,
                                                const int32_t input_size) {
  CHECK_LE(input_size, N) << input_size;
  const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
  const int64_t batch_size = dy->shape().At(0);
  const int64_t vector_size = dy->shape().At(1);
  int block_dim_x;
  int block_dim_y;
  GetBlockDims(vector_size, &block_dim_x, &block_dim_y);
  const int num_blocks = GetNumBlocks(batch_size, block_dim_y);
  dim3 block_dims = dim3(block_dim_x, block_dim_y);
  GradParam<T, N> param;
  param.num_in = input_size;
  param.dy = dy->dptr<T>();
  for (int i = 0; i < input_size; ++i) {
    param.in[i] = ctx->Tensor4ArgNameAndIndex("features", i)->dptr<T>();
    param.in_grad[i] = ctx->Tensor4ArgNameAndIndex("features_grad", i)->mut_dptr<T>();
    param.in_feature_dim[i] = ctx->TensorDesc4ArgNameAndIndex("features_grad", i)->shape().At(1);
  }
  FeatureInteractionSumGrad<T, N>
      <<<num_blocks, block_dims, 0, ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          batch_size, vector_size, param);
}

}  // namespace

template<typename T>
class FusedDotFeatureInteractionPoolingSumKernel final : public user_op::OpKernel,
                                                         public user_op::CudaGraphSupport {
 public:
  FusedDotFeatureInteractionPoolingSumKernel() = default;
  ~FusedDotFeatureInteractionPoolingSumKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const int input_size = ctx->input_size("features");
    if (input_size == 1) {
      DispatchFeatureInteractionSumInputSize<T, 1>(ctx, input_size);
    } else if (input_size == 2) {
      DispatchFeatureInteractionSumInputSize<T, 2>(ctx, input_size);
    } else if (input_size <= 8) {
      DispatchFeatureInteractionSumInputSize<T, 8>(ctx, input_size);
    } else {
      CHECK_LE(input_size, 128) << "input_size must not greater than 128. ";
      DispatchFeatureInteractionSumInputSize<T, 128>(ctx, input_size);
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_KERNEL(dtype)                \
  REGISTER_USER_KERNEL("fused_dot_feature_interaction")                                 \
      .SetCreateFn<FusedDotFeatureInteractionPoolingSumKernel<dtype>>()                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobAttr<std::string>("pooling") == "sum"));

REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_KERNEL(float)
REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_KERNEL(half)

template<typename T>
bool TryLaunchTensorCoreDotKernel(user_op::KernelComputeContext* ctx) {
  const int input_size = ctx->input_size("features");
  if (input_size == 1) {
    return DispatchFeatureInteractionDotPackSize<T, 1>(ctx, input_size);
  } else if (input_size == 2) {
    return DispatchFeatureInteractionDotPackSize<T, 2>(ctx, input_size);
  } else if (input_size <= 8) {
    return DispatchFeatureInteractionDotPackSize<T, 8>(ctx, input_size);
  } else {
    CHECK_LE(input_size, 128) << "input_size must not greater than 128. ";
    return DispatchFeatureInteractionDotPackSize<T, 128>(ctx, input_size);
  }
}

template<typename T>
bool TryLaunchTensorCoreDotBackwardKernel(user_op::KernelComputeContext* ctx) {
  const int input_size = ctx->input_size("features");
  if (input_size == 1) {
    return DispatchFeatureInteractionDotBackwardPackSize<T, 1>(ctx, input_size);
  } else if (input_size == 2) {
    return DispatchFeatureInteractionDotBackwardPackSize<T, 2>(ctx, input_size);
  } else if (input_size <= 8) {
    return DispatchFeatureInteractionDotBackwardPackSize<T, 8>(ctx, input_size);
  } else {
    CHECK_LE(input_size, 128) << "input_size must not greater than 128. ";
    return DispatchFeatureInteractionDotBackwardPackSize<T, 128>(ctx, input_size);
  }
}
template<typename T>
class FusedDotFeatureInteractionKernel final : public user_op::OpKernel,
                                               public user_op::CudaGraphSupport {
 public:
  FusedDotFeatureInteractionKernel() = default;
  ~FusedDotFeatureInteractionKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const DataType data_type = out->data_type();
    CHECK_LT(out->shape().elem_cnt(), GetMaxVal<int32_t>());
    auto* hip_stream = ctx->stream()->As<ep::CudaStream>();
    if (hip_stream->device_properties().major >= 7 && data_type == DataType::kFloat16) {
      bool success = TryLaunchTensorCoreDotKernel<T>(ctx);
      if (success == true) { return; }
    }
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t batch_size = out->shape().At(0);
    int64_t features_concated_dim = 0;
    for (int64_t i = 0; i < ctx->input_size("features"); ++i) {
      features_concated_dim += ctx->TensorDesc4ArgNameAndIndex("features", i)->shape().At(1);
    }
    const int64_t concated_padded_dim = GetPaddedDim(features_concated_dim);
    const int64_t vector_size = ctx->TensorDesc4ArgNameAndIndex("features", 0)->shape().At(2);
    const int64_t out_dim = out->shape().At(1);
    const int32_t output_padding = ctx->Attr<int32_t>("output_padding");
    const int64_t valid_out_dim = out_dim - output_padding;
    const bool self_interaction = ctx->Attr<bool>("self_interaction");

    T* matmul_out = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>());
    size_t matmul_out_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * concated_padded_dim * sizeof(T));
    const int64_t interaction_dim = self_interaction
                                        ? features_concated_dim * (features_concated_dim + 1) / 2
                                        : features_concated_dim * (features_concated_dim - 1) / 2;
    int32_t* gather_indices_ptr =
        reinterpret_cast<int32_t*>(tmp_buffer->mut_dptr<char>() + matmul_out_size);
    size_t gather_indices_size = GetCudaAlignedSize(interaction_dim * sizeof(int32_t));
    T* padded_concated_features_ptr =
        reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + matmul_out_size + gather_indices_size);
    size_t padded_concated_features_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * vector_size * sizeof(T));
    CHECK_GE(tmp_buffer->shape().elem_cnt(),
             matmul_out_size + gather_indices_size + padded_concated_features_size);
    ConcatFeatures<T>(ctx, batch_size, concated_padded_dim * vector_size,
                      padded_concated_features_ptr);
    auto batch_matmul = ep::primitive::NewPrimitive<ep::primitive::BatchMatmulFactory>(
        ctx->device_type(), data_type, ep::primitive::BlasTransposeType::N,
        ep::primitive::BlasTransposeType::T);
    batch_matmul->Launch(ctx->stream(), batch_size, concated_padded_dim, concated_padded_dim,
                         vector_size, 1.0, padded_concated_features_ptr,
                         padded_concated_features_ptr, 0.0, matmul_out);

    int64_t output_concat_end_dim = 0;
    const T* output_concat_ptr = nullptr;
    if (ctx->has_input("output_concat", 0)) {
      user_op::Tensor* output_concat = ctx->Tensor4ArgNameAndIndex("output_concat", 0);
      output_concat_end_dim = output_concat->shape().At(1);
      output_concat_ptr = output_concat->dptr<T>();
    }
    CHECK_EQ(valid_out_dim, output_concat_end_dim + interaction_dim);
    GatherConcatKernel<T>(ctx->stream(), out->shape().elem_cnt(), out_dim, valid_out_dim,
                          features_concated_dim, concated_padded_dim, output_concat_end_dim,
                          self_interaction, matmul_out, output_concat_ptr, gather_indices_ptr,
                          out->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
user_op::InferTmpSizeFn GenFusedDotFeatureInteractionInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    const Shape& first_feature_shape = ctx->InputShape("features", 0);
    const int64_t batch_size = first_feature_shape.At(0);
    const int64_t vector_size = first_feature_shape.At(2);
    int64_t features_concated_dim = 0;
    for (int32_t i = 0; i < ctx->input_size("features"); ++i) {
      features_concated_dim += ctx->InputShape("features", i).At(1);
    }
    const int64_t concated_padded_dim = GetPaddedDim(features_concated_dim);
    size_t matmul_out_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * concated_padded_dim * sizeof(T));
    const bool self_interaction = ctx->Attr<bool>("self_interaction");
    const int64_t interaction_dim = self_interaction
                                        ? features_concated_dim * (features_concated_dim + 1) / 2
                                        : features_concated_dim * (features_concated_dim - 1) / 2;
    size_t gather_indices_size = GetCudaAlignedSize(interaction_dim * sizeof(int32_t));
    size_t padded_concated_features_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * vector_size * sizeof(T));
    return matmul_out_size + gather_indices_size + padded_concated_features_size;
  };
}

#define REGISTER_FUSED_DOT_FEATURE_INTERACTION_KERNEL(dtype)                            \
  REGISTER_USER_KERNEL("fused_dot_feature_interaction")                                 \
      .SetCreateFn<FusedDotFeatureInteractionKernel<dtype>>()                           \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobAttr<std::string>("pooling") == "none"))         \
      .SetInferTmpSizeFn(GenFusedDotFeatureInteractionInferTmpSizeFn<dtype>());

REGISTER_FUSED_DOT_FEATURE_INTERACTION_KERNEL(float)
REGISTER_FUSED_DOT_FEATURE_INTERACTION_KERNEL(half)

template<typename T>
class FusedDotFeatureInteractionGradKernel final : public user_op::OpKernel,
                                                   public user_op::CudaGraphSupport {
 public:
  FusedDotFeatureInteractionGradKernel() = default;
  ~FusedDotFeatureInteractionGradKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const DataType data_type = dy->data_type();
    auto* hip_stream = ctx->stream()->As<ep::CudaStream>();
    if (hip_stream->device_properties().major >= 7 && data_type == DataType::kFloat16) {
      bool success = TryLaunchTensorCoreDotBackwardKernel<T>(ctx);
      if (success == true) { return; }
    }
    const int64_t batch_size = dy->shape().At(0);
    int64_t features_concated_dim = 0;
    for (int32_t i = 0; i < ctx->output_size("features_grad"); ++i) {
      features_concated_dim += ctx->TensorDesc4ArgNameAndIndex("features_grad", i)->shape().At(1);
    }
    const int64_t concated_padded_dim = GetPaddedDim(features_concated_dim);
    const int64_t vector_size = ctx->TensorDesc4ArgNameAndIndex("features_grad", 0)->shape().At(2);
    const int64_t out_dim = dy->shape().At(1);
    const bool self_interaction = ctx->Attr<bool>("self_interaction");
    T* matmul_out_grad_ptr = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>());
    size_t matmul_out_grad_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * concated_padded_dim * sizeof(T));
    T* padded_concated_features_grad_ptr =
        reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + matmul_out_grad_size);
    size_t padded_concated_features_grad_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * vector_size * sizeof(T));
    T* padded_concated_features_ptr = reinterpret_cast<T*>(
        tmp_buffer->mut_dptr<char>() + matmul_out_grad_size + padded_concated_features_grad_size);
    size_t padded_concated_features_size = padded_concated_features_grad_size;
    CHECK_LE(
        matmul_out_grad_size + padded_concated_features_grad_size + padded_concated_features_size,
        tmp_buffer->shape().elem_cnt());
    ConcatFeatures<T>(ctx, batch_size, concated_padded_dim * vector_size,
                      padded_concated_features_ptr);

    T* output_concat_grad_ptr = nullptr;
    int64_t output_concat_end_dim = 0;
    if (ctx->has_output("output_concat_grad", 0)) {
      user_op::Tensor* output_concat_grad = ctx->Tensor4ArgNameAndIndex("output_concat_grad", 0);
      output_concat_grad_ptr = output_concat_grad->mut_dptr<T>();
      output_concat_end_dim = output_concat_grad->shape().At(1);
    }
    ScatterSplitAddTranspose(ctx->stream(), batch_size, out_dim, concated_padded_dim,
                             features_concated_dim, output_concat_end_dim, self_interaction,
                             dy->dptr<T>(), output_concat_grad_ptr, matmul_out_grad_ptr);

    auto batch_matmul = ep::primitive::NewPrimitive<ep::primitive::BatchMatmulFactory>(
        ctx->device_type(), data_type, ep::primitive::BlasTransposeType::N,
        ep::primitive::BlasTransposeType::N);
    batch_matmul->Launch(ctx->stream(), batch_size, concated_padded_dim, vector_size,
                         concated_padded_dim, 1.0, matmul_out_grad_ptr,
                         padded_concated_features_ptr, 0.0, padded_concated_features_grad_ptr);

    ConcatFeaturesGrad(ctx, batch_size, concated_padded_dim, vector_size,
                       padded_concated_features_grad_ptr);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
user_op::InferTmpSizeFn GenFusedDotFeatureInteractionGradInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    int64_t features_concated_dim = 0;
    for (int32_t i = 0; i < ctx->output_size("features_grad"); ++i) {
      features_concated_dim += ctx->InputShape("features_grad", i).At(1);
    }
    const int64_t concated_padded_dim = GetPaddedDim(features_concated_dim);
    const int64_t batch_size = ctx->InputShape("features_grad", 0).At(0);
    const int64_t vector_size = ctx->InputShape("features_grad", 0).At(2);
    size_t matmul_out_grad_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * concated_padded_dim * sizeof(T));
    size_t padded_concated_features_grad_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * vector_size * sizeof(T));
    size_t padded_concated_features_size = padded_concated_features_grad_size;
    return matmul_out_grad_size + padded_concated_features_grad_size
           + padded_concated_features_size;
  };
}

#define REGISTER_FUSED_DOT_FEATURE_INTERACTION_GRAD_KERNEL(dtype)                      \
  REGISTER_USER_KERNEL("fused_dot_feature_interaction_grad")                           \
      .SetCreateFn<FusedDotFeatureInteractionGradKernel<dtype>>()                      \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                 \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobAttr<std::string>("pooling") == "none"))        \
      .SetInferTmpSizeFn(GenFusedDotFeatureInteractionGradInferTmpSizeFn<dtype>());

REGISTER_FUSED_DOT_FEATURE_INTERACTION_GRAD_KERNEL(float)
REGISTER_FUSED_DOT_FEATURE_INTERACTION_GRAD_KERNEL(half)

template<typename T>
class FusedDotFeatureInteractionPoolingSumGradKernel final : public user_op::OpKernel,
                                                             public user_op::CudaGraphSupport {
 public:
  FusedDotFeatureInteractionPoolingSumGradKernel() = default;
  ~FusedDotFeatureInteractionPoolingSumGradKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const int input_size = ctx->input_size("features");
    if (input_size == 1) {
      DispatchFeatureInteractionSumGradInputSize<T, 1>(ctx, input_size);
    } else if (input_size == 2) {
      DispatchFeatureInteractionSumGradInputSize<T, 2>(ctx, input_size);
    } else if (input_size <= 8) {
      DispatchFeatureInteractionSumGradInputSize<T, 8>(ctx, input_size);
    } else {
      CHECK_LE(input_size, 128) << "input_size must not greater than 128. ";
      DispatchFeatureInteractionSumGradInputSize<T, 128>(ctx, input_size);
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_GRAD_KERNEL(dtype)          \
  REGISTER_USER_KERNEL("fused_dot_feature_interaction_grad")                           \
      .SetCreateFn<FusedDotFeatureInteractionPoolingSumGradKernel<dtype>>()            \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                 \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobAttr<std::string>("pooling") == "sum"));

REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_GRAD_KERNEL(float)
REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_GRAD_KERNEL(half)

}  // namespace oneflow
