#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/ep/include/primitive/copy_nd.h"
#include "oneflow/core/ep/include/primitive/batch_matmul.h"
#include "oneflow/core/kernel/cuda_graph_support.h"

namespace oneflow {

namespace {

__global__ void GenerateGatherIndicesGpu(const int32_t elem_cnt, const int32_t stride,
                                         const int32_t in_cols, const int32_t offset,
                                         int32_t* gather_indices) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const int32_t row = i / stride;
    const int32_t col = i - row * stride;
    if (col < row + offset) {
      int32_t in_index = row * in_cols + col;
      int32_t idx = row * (offset + row - 1 + offset) / 2 + col;
      gather_indices[idx] = in_index;
    }
  }
}

template<typename T>
__global__ void GatherConcatGpu(int32_t elem_cnt, int32_t out_cols, int32_t valid_out_cols,
                                int32_t in_cols, int32_t output_concat_end_dim,
                                const int32_t* gather_indices, const T* in,
                                const T* output_concat_ptr, T* out_ptr) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const int32_t row = i / out_cols;
    const int32_t col = i - row * out_cols;
    T out_val;
    if (col < output_concat_end_dim) {
      const int32_t output_concat_idx = row * output_concat_end_dim + col;
      out_val = output_concat_ptr[output_concat_idx];
    } else if (col < valid_out_cols) {
      const int32_t gather_col_idx = gather_indices[col - output_concat_end_dim];
      const int32_t in_offset = row * in_cols + gather_col_idx;
      out_val = in[in_offset];
    } else {
      out_val = 0;
    }
    out_ptr[i] = out_val;
  }
}

template<typename T>
__global__ void ScatterSplitAddTransposeGpu(int32_t elem_cnt, int32_t stride_dim, int32_t out_dim,
                                            int32_t in_grad_stride, int32_t in_grad_matrix_dim,
                                            int32_t in_grad_matrix_valid_dim,
                                            int32_t output_concat_end_dim, const int32_t offset,
                                            const T* dy, T* output_concat_grad, T* in_grad) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const int32_t row = i / stride_dim;
    const int32_t col = i - row * stride_dim;
    if (col < output_concat_end_dim) {
      output_concat_grad[row * output_concat_end_dim + col] = dy[row * out_dim + col];
    } else {
      int32_t in_col_id = col - output_concat_end_dim;
      const int32_t matrix_row = in_col_id / in_grad_matrix_dim;
      const int32_t matrix_col = in_col_id - matrix_row * in_grad_matrix_dim;
      T grad_val = 0;
      const T* row_dy = dy + row * out_dim + output_concat_end_dim;
      if (matrix_row < in_grad_matrix_valid_dim && matrix_col < in_grad_matrix_valid_dim) {
        if (matrix_col < matrix_row) {
          int32_t dy_col_idx = matrix_row * (offset + matrix_row - 1 + offset) / 2 + matrix_col;
          grad_val = row_dy[dy_col_idx];
        } else if (matrix_row < matrix_col) {
          // transpose add
          int32_t trans_row_id = matrix_col;
          int32_t trans_col_id = matrix_row;
          int32_t dy_col_idx =
              trans_row_id * (offset + trans_row_id - 1 + offset) / 2 + trans_col_id;
          grad_val = row_dy[dy_col_idx];
        } else if ((matrix_row == matrix_col) && (offset == 1)) {
          int32_t dy_col_idx = matrix_row * (offset + matrix_row - 1 + offset) / 2 + matrix_col;
          grad_val = row_dy[dy_col_idx] * static_cast<T>(2);
        }
      }
      int32_t in_grad_offset = row * in_grad_stride + in_col_id;
      in_grad[in_grad_offset] = grad_val;
    }
  }
}

template<typename T>
void ConcatFeatures(user_op::KernelComputeContext* ctx) {
  const int64_t feature_input_size = ctx->input_size("features");
  user_op::Tensor* padded_concated_features =
      ctx->Tensor4ArgNameAndIndex("padded_concated_features", 0);
  auto primitive = ep::primitive::NewPrimitive<ep::primitive::CopyNdFactory>(DeviceType::kCUDA, 2);
  const int64_t dst_rows = padded_concated_features->shape().At(0);
  const int64_t dst_cols = padded_concated_features->shape().Count(1);
  void* dst_ptr = padded_concated_features->mut_dptr();
  DimVector dst_shape = {dst_rows, dst_cols};
  int64_t out_col_offset = 0;
  for (int64_t i = 0; i < feature_input_size; ++i) {
    const user_op::Tensor* feature = ctx->Tensor4ArgNameAndIndex("features", i);
    const int64_t feature_rows = feature->shape().At(0);
    const int64_t feature_cols = feature->shape().Count(1);
    DimVector dst_pos_vec = {0, out_col_offset};
    DimVector src_shape = {feature_rows, feature_cols};
    DimVector src_pos_vec = {0, 0};
    DimVector extent_vec = {feature_rows, feature_cols};
    primitive->Launch(ctx->stream(), feature->data_type(), 2, dst_ptr, dst_shape.data(),
                      dst_pos_vec.data(), feature->dptr<T>(), src_shape.data(), src_pos_vec.data(),
                      extent_vec.data());
    out_col_offset += feature_cols;
  }
  int64_t pad_dim = dst_cols - out_col_offset;
  if (pad_dim > 0) {
    char* out_ptr = reinterpret_cast<char*>(dst_ptr) + out_col_offset * sizeof(T);
    OF_CUDA_CHECK(hipMemset2DAsync(out_ptr, dst_cols * sizeof(T), 0, pad_dim * sizeof(T), dst_rows,
                                    ctx->stream()->As<ep::CudaStream>()->hip_stream()));
  }
}

template<typename T>
void GatherConcatKernel(ep::Stream* stream, int32_t elem_cnt, int32_t out_dim,
                        int32_t valid_out_dim, int32_t features_concated_dim,
                        int32_t concated_padded_dim, int32_t output_concat_end_dim,
                        bool self_interaction, const T* matmul_out, const T* output_concat_ptr,
                        int32_t* gather_indices_ptr, T* out_ptr) {
  hipStream_t hip_stream = stream->As<ep::CudaStream>()->hip_stream();
  const int32_t gen_indices_elem_cnt = features_concated_dim * features_concated_dim;
  int32_t offset = self_interaction ? 1 : 0;
  GenerateGatherIndicesGpu<<<BlocksNum4ThreadsNum(gen_indices_elem_cnt), kCudaThreadsNumPerBlock, 0,
                             hip_stream>>>(gen_indices_elem_cnt, features_concated_dim,
                                            concated_padded_dim, offset, gather_indices_ptr);

  int32_t matmul_stride = concated_padded_dim * concated_padded_dim;
  GatherConcatGpu<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, hip_stream>>>(
      elem_cnt, out_dim, valid_out_dim, matmul_stride, output_concat_end_dim, gather_indices_ptr,
      matmul_out, output_concat_ptr, out_ptr);
}

template<typename T>
void ScatterSplitAddTranspose(ep::Stream* stream, int32_t batch_size, int32_t out_dim,
                              int32_t concated_padded_dim, int32_t features_concated_dim,
                              int32_t output_concat_end_dim, const bool self_interaction,
                              const T* dy, T* output_concat_grad, T* matmul_out_grad_ptr) {
  int32_t stride_dim = output_concat_end_dim + concated_padded_dim * concated_padded_dim;
  int32_t matmul_stride = concated_padded_dim * concated_padded_dim;
  const int32_t elem_cnt = batch_size * stride_dim;
  int32_t offset = self_interaction ? 1 : 0;
  ScatterSplitAddTransposeGpu<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                stream->As<ep::CudaStream>()->hip_stream()>>>(
      elem_cnt, stride_dim, out_dim, matmul_stride, concated_padded_dim, features_concated_dim,
      output_concat_end_dim, offset, dy, output_concat_grad, matmul_out_grad_ptr);
}

template<typename T>
void ConcatFeaturesGrad(user_op::KernelComputeContext* ctx, const int64_t batch_size,
                        const int64_t concated_padded_dim, const int64_t vector_size,
                        const T* concated_features_grad) {
  auto primitive = ep::primitive::NewPrimitive<ep::primitive::CopyNdFactory>(DeviceType::kCUDA, 2);
  DimVector src_shape = {batch_size, concated_padded_dim * vector_size};
  int64_t in_col_offset = 0;
  for (int64_t i = 0; i < ctx->output_size("features_grad"); ++i) {
    user_op::Tensor* feature_grad = ctx->Tensor4ArgNameAndIndex("features_grad", i);
    const int64_t feature_grad_rows = feature_grad->shape().At(0);
    const int64_t feature_grad_cols = feature_grad->shape().Count(1);
    DimVector dst_shape = {feature_grad_rows, feature_grad_cols};
    DimVector dst_pos_vec = {0, 0};
    DimVector src_pos_vec = {0, in_col_offset};
    DimVector extent_vec = {feature_grad_rows, feature_grad_cols};
    in_col_offset += feature_grad_cols;
    primitive->Launch(ctx->stream(), feature_grad->data_type(), 2, feature_grad->mut_dptr(),
                      dst_shape.data(), dst_pos_vec.data(), concated_features_grad,
                      src_shape.data(), src_pos_vec.data(), extent_vec.data());
  }
}

template<typename T>
struct DefaultComputeType {
  using type = T;
};

template<>
struct DefaultComputeType<half> {
  using type = float;
};

template<typename T, size_t pack_size>
struct alignas(sizeof(T) * pack_size) Pack {
  T elem[pack_size];
};

template<typename T, int32_t N>
struct Param {
  const T* in[N];
  int32_t in_feature_dim[N];
  T* out;
  int32_t num_in;
};

template<typename T, int32_t N, int32_t pack_size>
__global__ void FeatureInteractionSum(int64_t batch_size, int64_t vector_num_pack,
                                      Param<T, N> param) {
  using ComputeType = typename DefaultComputeType<T>::type;
  Pack<T, pack_size>* dst_pack = reinterpret_cast<Pack<T, pack_size>*>(param.out);
  for (int batch_idx = blockIdx.x * blockDim.y + threadIdx.y; batch_idx < batch_size;
       batch_idx += gridDim.x * blockDim.y) {
    Pack<T, pack_size>* batch_out = dst_pack + batch_idx * vector_num_pack;
    for (int col_id = threadIdx.x; col_id < vector_num_pack; col_id += blockDim.x) {
      Pack<ComputeType, pack_size> sum;
      Pack<ComputeType, pack_size> square_sum;
#pragma unroll
      for (int k = 0; k < pack_size; ++k) {
        sum.elem[k] = static_cast<ComputeType>(0);
        square_sum.elem[k] = static_cast<ComputeType>(0);
      }
      for (int i = 0; i < N; ++i) {
        if (i >= param.num_in) { break; }
        const Pack<T, pack_size>* batch_in =
            reinterpret_cast<const Pack<T, pack_size>*>(param.in[i])
            + batch_idx * param.in_feature_dim[i] * vector_num_pack;
#pragma unroll
        for (int j = 0; j < param.in_feature_dim[i]; ++j) {
          Pack<T, pack_size> val = batch_in[j * vector_num_pack + col_id];
#pragma unroll
          for (int k = 0; k < pack_size; ++k) {
            const ComputeType compute_val = static_cast<ComputeType>(val.elem[k]);
            sum.elem[k] += compute_val;
            square_sum.elem[k] += compute_val * compute_val;
          }
        }
      }
      Pack<T, pack_size> out;
#pragma unroll
      for (int k = 0; k < pack_size; ++k) {
        out.elem[k] = static_cast<T>((sum.elem[k] * sum.elem[k] - square_sum.elem[k])
                                     * static_cast<ComputeType>(0.5));
      }
      batch_out[col_id] = out;
    }
  }
}

template<typename T, int32_t N>
struct GradParam {
  const T* dy;
  const T* in[N];
  int32_t in_feature_dim[N];
  T* in_grad[N];
  int32_t num_in;
};

template<typename T, int32_t N>
__global__ void FeatureInteractionSumGrad(int64_t batch_size, int64_t vector_size,
                                          GradParam<T, N> param) {
  using ComputeType = typename DefaultComputeType<T>::type;
  for (int batch_idx = blockIdx.x * blockDim.y + threadIdx.y; batch_idx < batch_size;
       batch_idx += gridDim.x * blockDim.y) {
    const T* batch_dy = param.dy + batch_idx * vector_size;
    for (int col_id = threadIdx.x; col_id < vector_size; col_id += blockDim.x) {
      ComputeType sum = 0;
      for (int i = 0; i < N; ++i) {
        if (i >= param.num_in) { break; }
        const T* batch_in = param.in[i] + batch_idx * param.in_feature_dim[i] * vector_size;
        for (int j = 0; j < param.in_feature_dim[i]; ++j) {
          sum += static_cast<ComputeType>(batch_in[j * vector_size + col_id]);
        }
      }
      for (int i = 0; i < N; ++i) {
        if (i >= param.num_in) { break; }
        const int64_t in_batch_offset = batch_idx * param.in_feature_dim[i] * vector_size;
        const T* batch_in = param.in[i] + in_batch_offset;
        T* batch_in_grad = param.in_grad[i] + in_batch_offset;
        for (int j = 0; j < param.in_feature_dim[i]; ++j) {
          const int64_t offset = j * vector_size + col_id;
          batch_in_grad[offset] =
              static_cast<T>(static_cast<ComputeType>(batch_dy[col_id])
                             * (sum - static_cast<ComputeType>(batch_in[offset])));
        }
      }
    }
  }
}

void GetBlockDims(const int64_t vector_size, int* block_dim_x, int* block_dim_y) {
  const int block_size = 256;
  if (vector_size < block_size) {
    *block_dim_x = std::ceil(static_cast<float>(vector_size) / 8) * 8;
    *block_dim_y = (block_size + *block_dim_x - 1) / *block_dim_x;
  } else {
    *block_dim_x = block_size;
    *block_dim_y = 1;
  }
}

int GetNumBlocks(const int64_t num_instances, const int64_t instance_per_block) {
  int max_blocks = (num_instances + instance_per_block - 1) / instance_per_block;
  return std::min(max_blocks, kCudaMaxBlocksNum);
}

template<typename T, int32_t N>
void DispatchFeatureInteractionSumPackSize(ep::Stream* stream, const int64_t batch_size,
                                           const int64_t vector_size, const Param<T, N>& param) {
  int block_dim_x;
  int block_dim_y;
  const int pack_size = (vector_size % 2 == 0) ? 2 : 1;
  const int64_t vector_num_pack = vector_size / pack_size;
  GetBlockDims(vector_num_pack, &block_dim_x, &block_dim_y);
  const int num_blocks = GetNumBlocks(batch_size, block_dim_y);
  dim3 block_dims = dim3(block_dim_x, block_dim_y);
  hipStream_t hip_stream = stream->As<ep::CudaStream>()->hip_stream();
  if (pack_size == 2) {
    FeatureInteractionSum<T, N, 2>
        <<<num_blocks, block_dims, 0, hip_stream>>>(batch_size, vector_num_pack, param);
  } else {
    FeatureInteractionSum<T, N, 1>
        <<<num_blocks, block_dims, 0, hip_stream>>>(batch_size, vector_num_pack, param);
  }
}

template<typename T, int N>
void DispatchFeatureInteractionSumInputSize(user_op::KernelComputeContext* ctx,
                                            const int32_t input_size) {
  CHECK_LE(input_size, N) << input_size;
  user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
  const int64_t batch_size = out->shape().At(0);
  const int64_t vector_size = out->shape().At(1);
  Param<T, N> param;
  param.num_in = input_size;
  param.out = out->mut_dptr<T>();
  for (int i = 0; i < input_size; ++i) {
    param.in[i] = ctx->Tensor4ArgNameAndIndex("features", i)->dptr<T>();
    param.in_feature_dim[i] = ctx->TensorDesc4ArgNameAndIndex("features", i)->shape().At(1);
  }
  DispatchFeatureInteractionSumPackSize<T, N>(ctx->stream(), batch_size, vector_size, param);
}

template<typename T, int N>
void DispatchFeatureInteractionSumGradInputSize(user_op::KernelComputeContext* ctx,
                                                const int32_t input_size) {
  CHECK_LE(input_size, N) << input_size;
  const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
  const int64_t batch_size = dy->shape().At(0);
  const int64_t vector_size = dy->shape().At(1);
  int block_dim_x;
  int block_dim_y;
  GetBlockDims(vector_size, &block_dim_x, &block_dim_y);
  const int num_blocks = GetNumBlocks(batch_size, block_dim_y);
  dim3 block_dims = dim3(block_dim_x, block_dim_y);
  GradParam<T, N> param;
  param.num_in = input_size;
  param.dy = dy->dptr<T>();
  for (int i = 0; i < input_size; ++i) {
    param.in[i] = ctx->Tensor4ArgNameAndIndex("features_grad_like", i)->dptr<T>();
    param.in_grad[i] = ctx->Tensor4ArgNameAndIndex("features_grad", i)->mut_dptr<T>();
    param.in_feature_dim[i] = ctx->TensorDesc4ArgNameAndIndex("features_grad", i)->shape().At(1);
  }
  FeatureInteractionSumGrad<T, N>
      <<<num_blocks, block_dims, 0, ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          batch_size, vector_size, param);
}

}  // namespace

template<typename T>
class FusedDotFeatureInteractionPoolingSumKernel final : public user_op::OpKernel,
                                                         public user_op::CudaGraphSupport {
 public:
  FusedDotFeatureInteractionPoolingSumKernel() = default;
  ~FusedDotFeatureInteractionPoolingSumKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const int input_size = ctx->input_size("features");
    if (input_size == 1) {
      DispatchFeatureInteractionSumInputSize<T, 1>(ctx, input_size);
    } else if (input_size == 2) {
      DispatchFeatureInteractionSumInputSize<T, 2>(ctx, input_size);
    } else if (input_size <= 8) {
      DispatchFeatureInteractionSumInputSize<T, 8>(ctx, input_size);
    } else {
      CHECK_LE(input_size, 128) << "input_size must not greater than 128. ";
      DispatchFeatureInteractionSumInputSize<T, 128>(ctx, input_size);
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_KERNEL(dtype)                \
  REGISTER_USER_KERNEL("fused_dot_feature_interaction")                                 \
      .SetCreateFn<FusedDotFeatureInteractionPoolingSumKernel<dtype>>()                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobAttr<std::string>("pooling") == "sum"));

REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_KERNEL(float)
REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_KERNEL(half)

template<typename T>
class FusedDotFeatureInteractionKernel final : public user_op::OpKernel,
                                               public user_op::CudaGraphSupport {
 public:
  FusedDotFeatureInteractionKernel() = default;
  ~FusedDotFeatureInteractionKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    CHECK_LT(out->shape().elem_cnt(), GetMaxVal<int32_t>());
    user_op::Tensor* padded_concated_features =
        ctx->Tensor4ArgNameAndIndex("padded_concated_features", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t batch_size = padded_concated_features->shape().At(0);
    int64_t features_concated_dim = 0;
    for (int64_t i = 0; i < ctx->input_size("features"); ++i) {
      features_concated_dim += ctx->TensorDesc4ArgNameAndIndex("features", i)->shape().At(1);
    }
    const int64_t concated_padded_dim = padded_concated_features->shape().At(1);
    const int64_t vector_size = padded_concated_features->shape().At(2);
    const int64_t out_dim = out->shape().At(1);
    const int32_t output_padding = ctx->Attr<int32_t>("output_padding");
    const int64_t valid_out_dim = out_dim - output_padding;
    const bool self_interaction = ctx->Attr<bool>("self_interaction");

    T* matmul_out = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>());
    size_t matmul_out_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * concated_padded_dim * sizeof(T));
    const int64_t interaction_dim = self_interaction
                                        ? features_concated_dim * (features_concated_dim + 1) / 2
                                        : features_concated_dim * (features_concated_dim - 1) / 2;
    int32_t* gather_indices_ptr =
        reinterpret_cast<int32_t*>(tmp_buffer->mut_dptr<char>() + matmul_out_size);

    ConcatFeatures<T>(ctx);
    auto batch_matmul = ep::primitive::NewPrimitive<ep::primitive::BatchMatmulFactory>(
        ctx->device_type(), padded_concated_features->data_type(),
        ep::primitive::BlasTransposeType::N, ep::primitive::BlasTransposeType::T);
    batch_matmul->Launch(ctx->stream(), batch_size, concated_padded_dim, concated_padded_dim,
                         vector_size, 1.0, padded_concated_features->dptr(),
                         padded_concated_features->dptr(), 0.0, matmul_out);

    int64_t output_concat_end_dim = 0;
    const T* output_concat_ptr = nullptr;
    if (ctx->has_input("output_concat", 0)) {
      user_op::Tensor* output_concat = ctx->Tensor4ArgNameAndIndex("output_concat", 0);
      output_concat_end_dim = output_concat->shape().At(1);
      output_concat_ptr = output_concat->dptr<T>();
    }
    CHECK_EQ(valid_out_dim, output_concat_end_dim + interaction_dim);
    GatherConcatKernel<T>(ctx->stream(), out->shape().elem_cnt(), out_dim, valid_out_dim,
                          features_concated_dim, concated_padded_dim, output_concat_end_dim,
                          self_interaction, matmul_out, output_concat_ptr, gather_indices_ptr,
                          out->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
user_op::InferTmpSizeFn GenFusedDotFeatureInteractionInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    const user_op::TensorDesc& padded_concated_features =
        ctx->InputTensorDesc("padded_concated_features", 0);
    const int64_t batch_size = padded_concated_features.shape().At(0);
    const int64_t vector_size = padded_concated_features.shape().At(2);
    int64_t features_concated_dim = 0;
    for (int32_t i = 0; i < ctx->input_size("features"); ++i) {
      features_concated_dim += ctx->InputTensorDesc("features", i).shape().At(1);
    }
    const int64_t concated_padded_dim = padded_concated_features.shape().At(1);
    const int64_t pad_dim = concated_padded_dim - features_concated_dim;
    size_t pad_tensor_size = GetCudaAlignedSize(batch_size * pad_dim * vector_size * sizeof(T));
    size_t matmul_out_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * concated_padded_dim * sizeof(T));
    const bool self_interaction = ctx->Attr<bool>("self_interaction");
    const int64_t interaction_dim = self_interaction
                                        ? features_concated_dim * (features_concated_dim + 1) / 2
                                        : features_concated_dim * (features_concated_dim - 1) / 2;
    size_t gather_indices_size = GetCudaAlignedSize(interaction_dim * sizeof(int32_t));
    return matmul_out_size + gather_indices_size + pad_tensor_size;
  };
}

#define REGISTER_FUSED_DOT_FEATURE_INTERACTION_KERNEL(dtype)                            \
  REGISTER_USER_KERNEL("fused_dot_feature_interaction")                                 \
      .SetCreateFn<FusedDotFeatureInteractionKernel<dtype>>()                           \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobAttr<std::string>("pooling") == "none"))         \
      .SetInferTmpSizeFn(GenFusedDotFeatureInteractionInferTmpSizeFn<dtype>());

REGISTER_FUSED_DOT_FEATURE_INTERACTION_KERNEL(float)
REGISTER_FUSED_DOT_FEATURE_INTERACTION_KERNEL(half)

template<typename T>
class FusedDotFeatureInteractionGradKernel final : public user_op::OpKernel,
                                                   public user_op::CudaGraphSupport {
 public:
  FusedDotFeatureInteractionGradKernel() = default;
  ~FusedDotFeatureInteractionGradKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* padded_concated_features =
        ctx->Tensor4ArgNameAndIndex("padded_concated_features", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const DataType data_type = dy->data_type();
    const int64_t batch_size = padded_concated_features->shape().At(0);
    int64_t features_concated_dim = 0;
    for (int32_t i = 0; i < ctx->output_size("features_grad"); ++i) {
      features_concated_dim += ctx->TensorDesc4ArgNameAndIndex("features_grad", i)->shape().At(1);
    }
    const int64_t concated_padded_dim = padded_concated_features->shape().At(1);
    const int64_t vector_size = padded_concated_features->shape().At(2);
    const int64_t out_dim = dy->shape().At(1);
    const bool self_interaction = ctx->Attr<bool>("self_interaction");
    T* matmul_out_grad_ptr = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>());
    size_t matmul_out_grad_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * concated_padded_dim * sizeof(T));
    T* padded_concated_features_grad_ptr =
        reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + matmul_out_grad_size);
    size_t padded_concated_features_grad_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * vector_size * sizeof(T));
    CHECK_LE(matmul_out_grad_size + padded_concated_features_grad_size,
             tmp_buffer->shape().elem_cnt());

    T* output_concat_grad_ptr = nullptr;
    int64_t output_concat_end_dim = 0;
    if (ctx->has_output("output_concat_grad", 0)) {
      user_op::Tensor* output_concat_grad = ctx->Tensor4ArgNameAndIndex("output_concat_grad", 0);
      output_concat_grad_ptr = output_concat_grad->mut_dptr<T>();
      output_concat_end_dim = output_concat_grad->shape().At(1);
    }
    ScatterSplitAddTranspose(ctx->stream(), batch_size, out_dim, concated_padded_dim,
                             features_concated_dim, output_concat_end_dim, self_interaction,
                             dy->dptr<T>(), output_concat_grad_ptr, matmul_out_grad_ptr);

    auto batch_matmul = ep::primitive::NewPrimitive<ep::primitive::BatchMatmulFactory>(
        ctx->device_type(), padded_concated_features->data_type(),
        ep::primitive::BlasTransposeType::N, ep::primitive::BlasTransposeType::N);
    batch_matmul->Launch(ctx->stream(), batch_size, concated_padded_dim, vector_size,
                         concated_padded_dim, 1.0, matmul_out_grad_ptr,
                         padded_concated_features->dptr(), 0.0, padded_concated_features_grad_ptr);

    ConcatFeaturesGrad(ctx, batch_size, concated_padded_dim, vector_size,
                       padded_concated_features_grad_ptr);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
user_op::InferTmpSizeFn GenFusedDotFeatureInteractionGradInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    const auto& padded_concated_features_shape =
        ctx->InputTensorDesc("padded_concated_features", 0).shape();
    const int64_t batch_size = padded_concated_features_shape.At(0);
    const int64_t concated_padded_dim = padded_concated_features_shape.At(1);
    const int64_t vector_size = padded_concated_features_shape.At(2);
    size_t matmul_out_grad_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * concated_padded_dim * sizeof(T));
    size_t padded_concated_features_grad_size =
        GetCudaAlignedSize(batch_size * concated_padded_dim * vector_size * sizeof(T));
    return matmul_out_grad_size + padded_concated_features_grad_size;
  };
}

#define REGISTER_FUSED_DOT_FEATURE_INTERACTION_GRAD_KERNEL(dtype)                      \
  REGISTER_USER_KERNEL("fused_dot_feature_interaction_grad")                           \
      .SetCreateFn<FusedDotFeatureInteractionGradKernel<dtype>>()                      \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                 \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobAttr<std::string>("pooling") == "none"))        \
      .SetInferTmpSizeFn(GenFusedDotFeatureInteractionGradInferTmpSizeFn<dtype>());

REGISTER_FUSED_DOT_FEATURE_INTERACTION_GRAD_KERNEL(float)
REGISTER_FUSED_DOT_FEATURE_INTERACTION_GRAD_KERNEL(half)

template<typename T>
class FusedDotFeatureInteractionPoolingSumGradKernel final : public user_op::OpKernel,
                                                             public user_op::CudaGraphSupport {
 public:
  FusedDotFeatureInteractionPoolingSumGradKernel() = default;
  ~FusedDotFeatureInteractionPoolingSumGradKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const int input_size = ctx->input_size("features_grad_like");
    if (input_size == 1) {
      DispatchFeatureInteractionSumGradInputSize<T, 1>(ctx, input_size);
    } else if (input_size == 2) {
      DispatchFeatureInteractionSumGradInputSize<T, 2>(ctx, input_size);
    } else if (input_size <= 8) {
      DispatchFeatureInteractionSumGradInputSize<T, 8>(ctx, input_size);
    } else {
      CHECK_LE(input_size, 128) << "input_size must not greater than 128. ";
      DispatchFeatureInteractionSumGradInputSize<T, 128>(ctx, input_size);
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_GRAD_KERNEL(dtype)          \
  REGISTER_USER_KERNEL("fused_dot_feature_interaction_grad")                           \
      .SetCreateFn<FusedDotFeatureInteractionPoolingSumGradKernel<dtype>>()            \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                 \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobAttr<std::string>("pooling") == "sum"));

REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_GRAD_KERNEL(float)
REGISTER_FUSED_DOT_FEATURE_INTERACTION_POOLING_SUM_GRAD_KERNEL(half)

}  // namespace oneflow
