#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/common/device_type.pb.h"
#include "oneflow/core/common/data_type_seq.h"
#include "oneflow/user/kernels/util_ops_kernels.h"
#include "oneflow/user/kernels/elementwise_xpu_kernel.cuh"

namespace oneflow {
namespace user_op {
#ifdef WITH_CUDA
template<typename T>
struct IsNanFunctor<DeviceType::kCUDA, T, std::enable_if_t<std::is_floating_point<T>::value>> {
  __device__ bool operator()(const T x) const { return isnan(x); }
};

template<typename T>
struct IsNanFunctor<DeviceType::kCUDA, T, std::enable_if_t<!std::is_floating_point<T>::value>> {
  __device__ bool operator()(const T x) const { return false; }
};

template<>
struct IsNanFunctor<DeviceType::kCUDA, half> {
  __device__ bool operator()(const half x) const {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
    return __hisnan(x);
#else
    return isnan(__half2float(x));
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__) */
  }
};

template<typename T>
struct IsInfFunctor<DeviceType::kCUDA, T, std::enable_if_t<std::is_floating_point<T>::value>> {
  __device__ bool operator()(const T x) const { return isinf(x); }
};

template<typename T>
struct IsInfFunctor<DeviceType::kCUDA, T, std::enable_if_t<!std::is_floating_point<T>::value>> {
  __device__ bool operator()(const T x) const { return false; }
};

template<>
struct IsInfFunctor<DeviceType::kCUDA, half> {
  __device__ bool operator()(const half x) const {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
    return __hisinf(x);
#else
    return isinf(__half2float(x));
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__) */
  }
};

#define REGISTER_UTIL_OPS_CUDA_KERNEL(device, dtype_pair)     \
  REGISTER_ISNAN_KERNEL(device, OF_PP_PAIR_FIRST(dtype_pair)) \
  REGISTER_ISINF_KERNEL(device, OF_PP_PAIR_FIRST(dtype_pair))

REGISTER_UTIL_OPS_CUDA_KERNEL(DeviceType::kCUDA, (half))
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_UTIL_OPS_CUDA_KERNEL, (DeviceType::kCUDA),
                                 UTIL_OPS_DATA_TYPE_SEQ);
#endif  // WITH_CUDA
}  // namespace user_op
}  // namespace oneflow
