#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"

namespace oneflow {

namespace {

template<typename T, typename K>
__global__ void OneHotEncodeGpu(int64_t elem_cnt, const int64_t depth, const T on_value,
                                const T off_value, const K* indices, T* out) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const int64_t row = i / depth;
    const int64_t col = i - row * depth;
    const int64_t idx = indices[row];
    assert(idx >= 0 && idx < depth);
    out[i] = (idx == col) ? on_value : off_value;
  }
}

}  // namespace

template<typename T, typename K>
class GpuOneHotKernel final : public user_op::OpKernel {
 public:
  GpuOneHotKernel() = default;
  ~GpuOneHotKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* indices = ctx->Tensor4ArgNameAndIndex("indices", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int64_t num_indices = indices->shape().elem_cnt();
    const int64_t depth = ctx->Attr<int64_t>("depth");
    const DataType dtype = ctx->Attr<DataType>("dtype");
    const T on_value = IsFloatingDataType(dtype)
                           ? static_cast<T>(ctx->Attr<double>("floating_on_value"))
                           : static_cast<T>(ctx->Attr<int64_t>("integer_on_value"));
    const T off_value = IsFloatingDataType(dtype)
                            ? static_cast<T>(ctx->Attr<double>("floating_off_value"))
                            : static_cast<T>(ctx->Attr<int64_t>("integer_off_value"));
    RUN_CUDA_KERNEL((OneHotEncodeGpu<T, K>), ctx->device_ctx(), num_indices * depth,
                    num_indices * depth, depth, on_value, off_value, indices->dptr<K>(),
                    out->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GPU_ONE_HOT_KERNEL(dtype, itype)                                               \
  REGISTER_USER_KERNEL("one_hot").SetCreateFn<GpuOneHotKernel<dtype, itype>>().SetIsMatchedHob( \
      (user_op::HobDeviceTag() == "gpu")                                                        \
      & (user_op::HobDataType("indices", 0) == GetDataType<itype>::value)                       \
      & (user_op::HobDataType("out", 0) == GetDataType<dtype>::value));

REGISTER_GPU_ONE_HOT_KERNEL(int32_t, int32_t)
REGISTER_GPU_ONE_HOT_KERNEL(int32_t, int64_t)
REGISTER_GPU_ONE_HOT_KERNEL(int64_t, int32_t)
REGISTER_GPU_ONE_HOT_KERNEL(int64_t, int64_t)
REGISTER_GPU_ONE_HOT_KERNEL(float, int32_t)
REGISTER_GPU_ONE_HOT_KERNEL(float, int64_t)
REGISTER_GPU_ONE_HOT_KERNEL(double, int32_t)
REGISTER_GPU_ONE_HOT_KERNEL(double, int64_t)

}  // namespace oneflow
