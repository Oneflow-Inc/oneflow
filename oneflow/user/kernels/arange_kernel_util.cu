
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#ifdef WITH_CUDA
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/arange_kernel_util.h"

namespace oneflow {

namespace user_op {

template<typename T>
__global__ void ArangeForwardGpuKernel(const T start, const T delta, const int64_t arange_elem_cnt,
                                       T* out) {
  // Use Loop to set the value
  DoArange<T>(start, delta, arange_elem_cnt, out);
}

template<>
__global__ void ArangeForwardGpuKernel(const half start, const half delta,
                                       const int64_t arange_elem_cnt, half* out) {
  // Use Loop to set the value
  XPU_1D_KERNEL_LOOP(i, arange_elem_cnt) {
    out[i] = start + static_cast<half>(static_cast<float>(i)) * delta;
  }
}

template<typename T>
struct ArangeFunctor<DeviceType::kCUDA, T> final {
  void operator()(ep::Stream* stream, const T start, const T delta, const int64_t arange_elem_cnt,
                  T* out) {
    // The thread num is set as arange_elem_cnt
    RUN_CUDA_KERNEL((ArangeForwardGpuKernel<T>), stream, arange_elem_cnt, start, delta,
                    arange_elem_cnt, out);
  }
};

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_ARANGE_FUNCTOR, (DeviceType::kCUDA),
                                 ARANGE_DATA_TYPE_SEQ);
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_ARANGE_FUNCTOR, (DeviceType::kCUDA),
                                 HALF_DATA_TYPE_SEQ);
}  // namespace user_op
}  // namespace oneflow

#endif  // End WITH_CUDA
