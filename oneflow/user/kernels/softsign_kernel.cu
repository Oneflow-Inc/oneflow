#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/kernel/util/cuda_half_util.h"
#include "oneflow/core/cuda/elementwise.cuh"
namespace oneflow {

namespace user_op {

template<typename T>
struct SoftsignFunctor {
  OF_DEVICE_FUNC T operator()(T x) const {
    return x / (static_cast<T>(1) + static_cast<T>(fabs(x)));
  }
};

template<typename T>
struct SoftsignGradFunctor {
  OF_DEVICE_FUNC T operator()(T x, T dy) const {
    return static_cast<T>(1.0) / (static_cast<T>(1.0) + static_cast<T>(fabs(x))) / (static_cast<T>(1.0) + static_cast<T>(fabs(x))) * dy;
  }
};

template<>
struct SoftsignFunctor<half> {
  SoftsignFunctor<float> float_functor;
  OF_DEVICE_FUNC half operator()(half x) const {
    return __float2half(float_functor(__half2float(x)));
  }
};

template<>
struct SoftsignGradFunctor<half> {
  SoftsignGradFunctor<float> float_functor;
  OF_DEVICE_FUNC half operator()(half x, half dy) const {
    return __float2half(float_functor(__half2float(x), __half2float(dy)));
  }
};

template<DeviceType device_type, typename T>
class GpuSoftsignKernel final : public OpKernel {
 public:
  GpuSoftsignKernel() = default;
  ~GpuSoftsignKernel() = default;

 private:
  void Compute(KernelComputeContext* ctx) const override {
    const Tensor* in_tensor = ctx->Tensor4ArgNameAndIndex("in", 0);
    Tensor* out_tensor = ctx->Tensor4ArgNameAndIndex("out", 0);
    const T* in_ptr = in_tensor->dptr<T>();
    T* out_ptr = out_tensor->mut_dptr<T>();
    const int32_t elem_cnt = in_tensor->shape().elem_cnt();
    OF_CUDA_CHECK((oneflow::cuda::elementwise::Unary(SoftsignFunctor<T>(), elem_cnt, out_ptr,
                                                     in_ptr, ctx->device_ctx()->hip_stream())));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GPU_SOFTSIGN_KERNEL(device, dtype)    \
  REGISTER_USER_KERNEL("softsign")                     \
      .SetCreateFn<GpuSoftsignKernel<device, dtype>>() \
      .SetIsMatchedHob((HobDeviceTag() == device)         \
                       & (HobDataType("out", 0) == GetDataType<dtype>::value));

REGISTER_GPU_SOFTSIGN_KERNEL(DeviceType::kGPU, half);
REGISTER_GPU_SOFTSIGN_KERNEL(DeviceType::kGPU, float);
REGISTER_GPU_SOFTSIGN_KERNEL(DeviceType::kGPU, double);

template<DeviceType device_type, typename T>
class GpuSoftsignGradKernel final : public OpKernel {
 public:
  GpuSoftsignGradKernel() = default;
  ~GpuSoftsignGradKernel() = default;

 private:
  void Compute(KernelComputeContext* ctx) const override {
    const Tensor* x_tensor = ctx->Tensor4ArgNameAndIndex("x", 0);
    const Tensor* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    Tensor* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const T* x_ptr = x_tensor->dptr<T>();
    const T* dy_ptr = dy_tensor->dptr<T>();
    T* dx_ptr = dx_tensor->mut_dptr<T>();

    const int32_t elem_cnt = x_tensor->shape().elem_cnt();
    OF_CUDA_CHECK(
        (oneflow::cuda::elementwise::Binary(SoftsignGradFunctor<T>(), elem_cnt, dx_ptr, x_ptr,
                                            dy_ptr, ctx->device_ctx()->hip_stream())));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GPU_SOFTSIGN_BACKWARD_KERNEL(device, dtype) \
  REGISTER_USER_KERNEL("softsign_grad")                      \
      .SetCreateFn<GpuSoftsignGradKernel<device, dtype>>()   \
      .SetIsMatchedHob((HobDeviceTag() == device)               \
                       & (HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_GPU_SOFTSIGN_BACKWARD_KERNEL(DeviceType::kGPU, half);
REGISTER_GPU_SOFTSIGN_BACKWARD_KERNEL(DeviceType::kGPU, float);
REGISTER_GPU_SOFTSIGN_BACKWARD_KERNEL(DeviceType::kGPU, double);

}  // namespace user_op

}  // namespace oneflow
