#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/embedding/key_value_store.h"
#include "oneflow/core/embedding/embedding_manager.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/user/kernels/random_mask_generator.h"
#include "oneflow/core/framework/random_generator_impl.h"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/core/ep/include/primitive/copy_nd.h"
#include "oneflow/core/ep/include/primitive/cast.h"
#include "oneflow/core/ep/include/device.h"

namespace oneflow {

namespace {

enum class InitializerType { kUniform, kNormal, kConstant };

struct EmbeddingInitializer {
  InitializerType type;
  union {
    struct {
      float low;
      float high;
    } uniform_param;
    struct {
      float mean;
      float std;
    } normal_param;
    struct {
      float value;
    } constant_param;
  };

  bool operator==(const EmbeddingInitializer& rhs) const {
    if (this->type != rhs.type) { return false; }
    if (rhs.type == InitializerType::kUniform) {
      return (this->uniform_param.low == rhs.uniform_param.low)
             && (this->uniform_param.high == rhs.uniform_param.high);
    } else if (rhs.type == InitializerType::kNormal) {
      return (this->normal_param.mean == rhs.normal_param.mean)
             && (this->normal_param.std == rhs.normal_param.std);
    } else if (rhs.type == InitializerType::kConstant) {
      return this->constant_param.value == rhs.constant_param.value;
    } else {
      UNIMPLEMENTED();
      return false;
    }
  }
};

void ParseInitializerFromJson(const nlohmann::json& initializer,
                              EmbeddingInitializer* embedding_initializer) {
  CHECK(initializer.contains("type"));
  CHECK(initializer["type"].is_string());
  std::string type = initializer["type"].get<std::string>();
  if (type == "uniform") {
    embedding_initializer->type = InitializerType::kUniform;
    CHECK(initializer.contains("low"));
    CHECK(initializer.contains("high"));
    CHECK(initializer["low"].is_number());
    CHECK(initializer["high"].is_number());
    embedding_initializer->uniform_param.low = initializer["low"];
    embedding_initializer->uniform_param.high = initializer["high"];
  } else if (type == "normal") {
    CHECK(initializer.contains("mean"));
    CHECK(initializer.contains("std"));
    CHECK(initializer["mean"].is_number());
    CHECK(initializer["std"].is_number());
    embedding_initializer->type = InitializerType::kNormal;
    embedding_initializer->normal_param.mean = initializer["mean"];
    embedding_initializer->normal_param.std = initializer["std"];
  } else if (type == "constant") {
    CHECK(initializer.contains("value"));
    CHECK(initializer["value"].is_number());
    embedding_initializer->type = InitializerType::kConstant;
    embedding_initializer->constant_param.value = initializer["value"];
  } else {
    UNIMPLEMENTED() << "Unsupported initializer type";
  }
}

int32_t ParseJsonToUniqueInitializerVecAndReturnOffset(
    const nlohmann::json& initializer, std::vector<EmbeddingInitializer>* initializers) {
  EmbeddingInitializer embedding_initializer;
  ParseInitializerFromJson(initializer, &embedding_initializer);
  for (int32_t i = 0; i < initializers->size(); ++i) {
    if (initializers->at(i) == embedding_initializer) { return i; }
  }
  initializers->push_back(embedding_initializer);
  return initializers->size() - 1;
}

void SetInitializerIndex(int32_t row_id, int32_t col_start, int32_t col_end, int64_t line_size,
                         int8_t index, std::vector<int8_t>* initializer_index) {
  int64_t row_offset = row_id * line_size;
  for (int32_t col = col_start; col < col_end; ++col) {
    initializer_index->at(row_offset + col) = index;
  }
}

void ParseAndSetStateInitializerIndex(const std::string& state_initializer,
                                      const int32_t num_tables, const int64_t line_size,
                                      const int64_t embedding_size,
                                      std::vector<EmbeddingInitializer>* initializer_params,
                                      std::vector<int8_t>* initializer_index) {
  if (line_size == embedding_size) { return; }
  CHECK(!state_initializer.empty());
  auto initializers = nlohmann::json::parse(state_initializer);
  CHECK(initializers.is_array());
  const int num_states = line_size / embedding_size - 1;
  CHECK_EQ(num_states, initializers.size());
  for (int32_t i = 0; i < num_states; ++i) {
    int32_t offset =
        ParseJsonToUniqueInitializerVecAndReturnOffset(initializers.at(i), initializer_params);
    int32_t col_start = embedding_size + i * embedding_size;
    int32_t col_end = col_start + embedding_size;
    CHECK_LE(col_end, line_size);
    for (int32_t j = 0; j < num_tables; ++j) {
      SetInitializerIndex(j, col_start, col_end, line_size, offset, initializer_index);
    }
  }
}

void ParseAndSetModelInitializerIndex(const nlohmann::json& tables,
                                      const std::vector<int64_t>& column_dims,
                                      const int32_t num_tables, const int32_t num_columns,
                                      const int64_t line_size, const int64_t embedding_size,
                                      std::vector<EmbeddingInitializer>* initializer_params,
                                      std::vector<int8_t>* initializer_index) {
  for (int32_t i = 0; i < num_tables; ++i) {
    auto table = tables.at(i);
    CHECK(table.contains("columns"));
    auto columns = table["columns"];
    CHECK(columns.is_array());
    CHECK_EQ(num_columns, columns.size()) << "columns size must equal to num embedding dims";
    int32_t col_start = 0;
    for (int k = 0; k < columns.size(); ++k) {
      auto column = columns.at(k);
      CHECK(column.contains("initializer"));
      int32_t offset =
          ParseJsonToUniqueInitializerVecAndReturnOffset(column["initializer"], initializer_params);
      int32_t col_end = col_start + column_dims.at(k);
      SetInitializerIndex(i, col_start, col_end, line_size, offset, initializer_index);
      col_start = col_end;
    }
    CHECK_EQ(col_start, embedding_size);
  }
}

void ParseInitializers(const int64_t line_size, const int64_t embedding_size,
                       const std::string& state_initializer, const std::string& json_serialized,
                       std::vector<EmbeddingInitializer>* initializer_params,
                       std::vector<int8_t>* initializer_index) {
  auto json_object = nlohmann::json::parse(json_serialized);
  CHECK(json_object.contains("column_dims"));
  std::vector<int64_t> column_dims = json_object["column_dims"];
  const int32_t num_columns = column_dims.size();
  CHECK(json_object.contains("tables"));
  auto tables = json_object["tables"];
  CHECK(tables.is_array());
  const int32_t num_tables = tables.size();
  initializer_index->resize(num_tables * line_size);
  ParseAndSetStateInitializerIndex(state_initializer, num_tables, line_size, embedding_size,
                                   initializer_params, initializer_index);
  ParseAndSetModelInitializerIndex(tables, column_dims, num_tables, num_columns, line_size,
                                   embedding_size, initializer_params, initializer_index);
}

template<typename IDX>
class EmbeddingKernelState final : public user_op::OpKernelState {
 public:
  explicit EmbeddingKernelState(user_op::KernelInitContext* ctx)
      : device_index_(-1), generator_(CHECK_JUST(one::MakeGenerator(DeviceType::kCUDA))) {
    OF_CUDA_CHECK(hipGetDevice(&device_index_));
    OF_CUDA_CHECK(hipHostMalloc(&host_num_keys_, sizeof(IDX)));
    key_value_store_ = Singleton<embedding::EmbeddingManager>::Get()->GetKeyValueStore(
        ctx->Attr<std::string>("embedding_name"), ctx->parallel_ctx().parallel_id());
    uint32_t max_query_length =
        ctx->TensorDesc4ArgNameAndIndex("unique_ids", 0)->shape().elem_cnt();
    key_value_store_->ReserveQueryLength(max_query_length);

    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const std::string& state_initializer = ctx->Attr<std::string>("state_initializer");

    std::vector<EmbeddingInitializer> initializer_param;
    std::vector<int8_t> initializer_index;
    ParseInitializers(line_size, embedding_size, state_initializer,
                      ctx->Attr<std::string>("embedding_tables"), &initializer_param,
                      &initializer_index);

    const size_t param_size_bytes = initializer_param.size() * sizeof(EmbeddingInitializer);
    OF_CUDA_CHECK(hipHostMalloc(&host_initializer_param_, param_size_bytes));
    std::memcpy(host_initializer_param_, initializer_param.data(), param_size_bytes);
    OF_CUDA_CHECK(hipMalloc(&device_initializer_param_, param_size_bytes));
    OF_CUDA_CHECK(hipMemcpyAsync(device_initializer_param_, host_initializer_param_,
                                  param_size_bytes, hipMemcpyDefault,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));

    const size_t index_size_bytes = initializer_index.size() * sizeof(int8_t);
    OF_CUDA_CHECK(hipHostMalloc(&host_initializer_index_, index_size_bytes));
    std::memcpy(host_initializer_index_, initializer_index.data(), index_size_bytes);
    OF_CUDA_CHECK(hipMalloc(&device_initializer_index_, index_size_bytes));
    OF_CUDA_CHECK(hipMemcpyAsync(device_initializer_index_, host_initializer_index_,
                                  index_size_bytes, hipMemcpyDefault,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));
  }
  ~EmbeddingKernelState() override {
    CudaCurrentDeviceGuard guard(device_index_);
    OF_CUDA_CHECK(hipHostFree(host_num_keys_));
    OF_CUDA_CHECK(hipHostFree(host_initializer_param_));
    OF_CUDA_CHECK(hipFree(device_initializer_param_));
    OF_CUDA_CHECK(hipHostFree(host_initializer_index_));
    OF_CUDA_CHECK(hipFree(device_initializer_index_));
  }

  void* HostNumKeys() { return host_num_keys_; }

  embedding::KeyValueStore* KeyValueStore() { return key_value_store_; }

  one::Generator* generator() { return generator_.get(); }

  const int8_t* InitializerIndex() { return device_initializer_index_; }
  const EmbeddingInitializer* Initializers() { return device_initializer_param_; }

 private:
  int device_index_;
  void* host_num_keys_;
  std::shared_ptr<one::Generator> generator_;
  embedding::KeyValueStore* key_value_store_;

  EmbeddingInitializer* host_initializer_param_;
  EmbeddingInitializer* device_initializer_param_;
  int8_t* host_initializer_index_;
  int8_t* device_initializer_index_;
};

template<typename IDX>
class EmbeddingPutKernelState final : public user_op::OpKernelState {
 public:
  explicit EmbeddingPutKernelState(user_op::KernelInitContext* ctx) : device_index_(-1) {
    OF_CUDA_CHECK(hipGetDevice(&device_index_));
    OF_CUDA_CHECK(hipHostMalloc(&host_num_keys_, sizeof(IDX)));
    key_value_store_ = Singleton<embedding::EmbeddingManager>::Get()->GetKeyValueStore(
        ctx->Attr<std::string>("embedding_name"), ctx->parallel_ctx().parallel_id());
    uint32_t max_query_length =
        ctx->TensorDesc4ArgNameAndIndex("unique_ids", 0)->shape().elem_cnt();
    key_value_store_->ReserveQueryLength(max_query_length);
  }
  ~EmbeddingPutKernelState() override {
    CudaCurrentDeviceGuard guard(device_index_);
    OF_CUDA_CHECK(hipHostFree(host_num_keys_));
  }

  void* HostNumKeys() { return host_num_keys_; }
  embedding::KeyValueStore* KeyValueStore() { return key_value_store_; }

 private:
  int device_index_;
  void* host_num_keys_;
  embedding::KeyValueStore* key_value_store_;
};

enum class EmbeddingBufferType { kNumMissing = 0, kMissingIndices, kValues, kMaxType };

class EmbeddingTmpBufferManager final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(EmbeddingTmpBufferManager);
  EmbeddingTmpBufferManager(void* ptr, const int64_t num_ids, const int64_t value_byte_size,
                            const bool need_value_buffer)
      : offset_(0), offsets_(static_cast<size_t>(EmbeddingBufferType::kMaxType), -1), ptr_(ptr) {
    AllocBuffer(EmbeddingBufferType::kNumMissing, sizeof(uint32_t));
    AllocBuffer(EmbeddingBufferType::kMissingIndices, num_ids * sizeof(uint32_t));
    if (need_value_buffer) { AllocBuffer(EmbeddingBufferType::kValues, num_ids * value_byte_size); }
  }

  template<typename T = void>
  T* Ptr(EmbeddingBufferType type) {
    CHECK(ptr_ != nullptr);
    int64_t offset = offsets_.at(static_cast<size_t>(type));
    CHECK_NE(offset, -1);
    return reinterpret_cast<T*>(reinterpret_cast<char*>(ptr_) + offset);
  }

  size_t TotalBufferSize() const { return offset_; }

 private:
  void AllocBuffer(EmbeddingBufferType type, size_t size) {
    const size_t type_id = static_cast<size_t>(type);
    CHECK_EQ(offsets_.at(type_id), -1);
    offsets_.at(type_id) = offset_;
    offset_ += GetCudaAlignedSize(size);
  }

  size_t offset_;
  std::vector<int64_t> offsets_;
  void* ptr_;
};

template<typename T, typename U>
__global__ void InitValueKernel(uint64_t seed, one::CUDAGeneratorState* cuda_gen_state,
                                uint64_t inc_offset, const int32_t line_size,
                                const int32_t embedding_size,
                                const EmbeddingInitializer* initializer_param,
                                const int8_t* initializer_index, const U* table_ids,
                                const uint32_t* num_missing_keys, const uint32_t* missing_indices,
                                T* values) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, cuda_gen_state->dev_offset, &state);
  int64_t n = *num_missing_keys * line_size;
  CUDA_1D_KERNEL_LOOP(i, n) {
    int row = i / line_size;
    int col = i - row * line_size;
    const uint32_t index = missing_indices[row];
    const int64_t offset = index * line_size + col;
    const int32_t table_idx = table_ids[index];
    const int32_t initializer_idx = initializer_index[table_idx * line_size + col];
    EmbeddingInitializer initializer = initializer_param[initializer_idx];
    T value;
    if (initializer.type == InitializerType::kUniform) {
      const float low = initializer.uniform_param.low;
      const float high = initializer.uniform_param.high;
      value = hiprand_uniform(&state) * (high - low) + low;
    } else if (initializer.type == InitializerType::kNormal) {
      const float mean = initializer.normal_param.mean;
      const float std = initializer.normal_param.std;
      value = hiprand_normal(&state) * std + mean;
    } else if (initializer.type == InitializerType::kConstant) {
      value = initializer.constant_param.value;
    } else {
      __trap();
    }
    values[offset] = value;
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    int32_t new_counter = cuda::atomic::Add(&cuda_gen_state->dev_counter, 1) + 1;
    if (new_counter == gridDim.x) {
      cuda_gen_state->dev_counter = 0;           // reset counter to zero
      cuda_gen_state->dev_offset += inc_offset;  // maintain the state of generator's dev_offset
    }
  }
}

template<typename T, typename U, typename IDX>
void LookupAndInitMissing(ep::Stream* stream, EmbeddingKernelState<IDX>* embedding_state,
                          const int64_t num_ids, const int64_t embedding_size,
                          const int64_t line_size, const void* num_unique_ptr,
                          const void* unique_ids, const void* table_ids, T* values_ptr,
                          void* tmp_buffer_ptr, uint32_t* return_num_unique,
                          const bool put_to_kv_store) {
  const auto& generator = embedding_state->generator();
  CHECK_NOTNULL(generator);
  std::shared_ptr<one::CUDAGeneratorImpl> cuda_generator =
      CHECK_JUST(generator->template Get<one::CUDAGeneratorImpl>(stream->device()->device_index()));
  uint64_t seed = cuda_generator->current_seed();
  one::CUDAGeneratorState* cuda_gen_state = cuda_generator->cuda_gen_state();
  embedding::KeyValueStore* store = embedding_state->KeyValueStore();
  const EmbeddingInitializer* initializer_param = embedding_state->Initializers();
  const int8_t* initializer_index = embedding_state->InitializerIndex();
  bool need_value_buffer = (values_ptr == nullptr);
  EmbeddingTmpBufferManager buffer_manager(tmp_buffer_ptr, num_ids, line_size * sizeof(T),
                                           need_value_buffer);
  void* host_num_keys = embedding_state->HostNumKeys();
  OF_CUDA_CHECK(hipMemcpyAsync(host_num_keys, num_unique_ptr, sizeof(IDX), hipMemcpyDefault,
                                stream->As<ep::CudaStream>()->hip_stream()));
  CHECK_JUST(stream->Sync());
  uint32_t num_unique = *reinterpret_cast<IDX*>(host_num_keys);
  uint32_t* num_missing_ptr =
      buffer_manager.template Ptr<uint32_t>(EmbeddingBufferType::kNumMissing);
  uint32_t* missing_indices =
      buffer_manager.template Ptr<uint32_t>(EmbeddingBufferType::kMissingIndices);
  T* store_values =
      need_value_buffer ? buffer_manager.template Ptr<T>(EmbeddingBufferType::kValues) : values_ptr;
  store->Get(stream, num_unique, unique_ids, store_values, num_missing_ptr, missing_indices);
  CHECK_GE(sizeof(IDX), sizeof(uint32_t));  // host_num_keys's buffer size is sizeof(IDX)
  OF_CUDA_CHECK(hipMemcpyAsync(host_num_keys, num_missing_ptr, sizeof(uint32_t), hipMemcpyDefault,
                                stream->As<ep::CudaStream>()->hip_stream()));
  CHECK_JUST(stream->Sync());
  uint32_t num_missing = *reinterpret_cast<uint32_t*>(host_num_keys);
  // init missing values
  if (num_missing > 0) {
    const int64_t elem_cnt = num_missing * line_size;
    const int64_t num_blocks = BlocksNum4ThreadsNum(elem_cnt);
    const uint64_t inc_offset = std::ceil(elem_cnt / num_blocks / kCudaThreadsNumPerBlock);
    InitValueKernel<T, U>
        <<<num_blocks, kCudaThreadsNumPerBlock, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            seed, cuda_gen_state, inc_offset, line_size, embedding_size, initializer_param,
            initializer_index, reinterpret_cast<const U*>(table_ids), num_missing_ptr,
            missing_indices, store_values);
  }
  if (put_to_kv_store) { store->Put(stream, num_unique, unique_ids, store_values); }
  *return_num_unique = num_unique;
}

template<typename T, typename U>
__global__ void Copy2D(int64_t out_elem_cnt, const int32_t in_cols, const int32_t out_cols,
                       const T* in, U* out) {
  CUDA_1D_KERNEL_LOOP(i, out_elem_cnt) {
    const int32_t row = i / out_cols;
    const int32_t col = i - row * out_cols;
    const int64_t in_offset = row * in_cols + col;
    out[i] = static_cast<U>(in[in_offset]);
  }
}

template<typename T>
void CopyValuesToEmbeddings(ep::Stream* stream, int64_t num_unique, const int32_t embedding_size,
                            const int32_t value_size, const DataType value_dtype,
                            const DataType embedding_dtype, const T* values, void* embeddings) {
  bool need_cast = (value_dtype != embedding_dtype);
  bool need_copy_nd = (embedding_size != value_size);
  CHECK(need_cast || need_copy_nd);
  if (need_cast && !need_copy_nd) {
    const int64_t cast_elem_count = num_unique * embedding_size;
    std::unique_ptr<ep::primitive::Cast> cast_primitive =
        ep::primitive::NewPrimitive<ep::primitive::CastFactory>(DeviceType::kCUDA, value_dtype,
                                                                embedding_dtype);
    cast_primitive->Launch(stream, values, embeddings, cast_elem_count);
  } else if (!need_cast && need_copy_nd) {
    const int32_t ndims = 2;
    DimVector src_pos_vec(ndims, 0);
    DimVector dst_pos_vec(ndims, 0);
    DimVector src_shape = {num_unique, value_size};
    DimVector dst_shape = {num_unique, embedding_size};
    DimVector extent_shape = {num_unique, embedding_size};
    std::unique_ptr<ep::primitive::CopyNd> copy_nd_primitive =
        ep::primitive::NewPrimitive<ep::primitive::CopyNdFactory>(DeviceType::kCUDA, ndims);
    CHECK(copy_nd_primitive);
    copy_nd_primitive->Launch(stream, value_dtype, ndims, embeddings, dst_shape.data(),
                              dst_pos_vec.data(), values, src_shape.data(), src_pos_vec.data(),
                              extent_shape.data());
  } else {
    const int64_t embedding_elem_cnt = num_unique * embedding_size;
    if (embedding_dtype == DataType::kFloat16) {
      Copy2D<T, half><<<BlocksNum4ThreadsNum(embedding_elem_cnt), kCudaThreadsNumPerBlock, 0,
                        stream->As<ep::CudaStream>()->hip_stream()>>>(
          embedding_elem_cnt, value_size, embedding_size, values,
          reinterpret_cast<half*>(embeddings));
    } else {
      UNIMPLEMENTED();
    }
  }
}

}  // namespace

template<typename T, typename U, typename IDX>
class EmbeddingPrefetchKernel final : public user_op::OpKernel {
 public:
  EmbeddingPrefetchKernel() = default;
  ~EmbeddingPrefetchKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingKernelState<IDX>>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* embedding_state = dynamic_cast<EmbeddingKernelState<IDX>*>(state);
    CHECK(embedding_state != nullptr);

    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* table_ids = ctx->Tensor4ArgNameAndIndex("table_ids", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    uint32_t num_unique;
    T* values_ptr = nullptr;
    LookupAndInitMissing<T, U, IDX>(ctx->stream(), embedding_state,
                                    unique_ids->shape_view().elem_cnt(), embedding_size, line_size,
                                    num_unique_ids->dptr(), unique_ids->dptr(), table_ids->dptr(),
                                    values_ptr, tmp_buffer->mut_dptr(), &num_unique, true);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define EMBEDDING_DATA_TYPE_SEQ OF_PP_MAKE_TUPLE_SEQ(float, DataType::kFloat)

#define TABLE_ID_DATA_TYPE_SEQ                      \
  OF_PP_MAKE_TUPLE_SEQ(uint8_t, DataType::kUInt8)   \
  OF_PP_MAKE_TUPLE_SEQ(uint32_t, DataType::kUInt32) \
  OF_PP_MAKE_TUPLE_SEQ(uint64_t, DataType::kUInt64) \
  OF_PP_MAKE_TUPLE_SEQ(int8_t, DataType::kInt8)     \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32)   \
  OF_PP_MAKE_TUPLE_SEQ(int64_t, DataType::kInt64)

#define IDX_DATA_TYPE_SEQ                           \
  OF_PP_MAKE_TUPLE_SEQ(uint32_t, DataType::kUInt32) \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32)

#define REGISTER_CUDA_EMBEDDING_PREFETCH_KERNEL(t_dtype_pair, table_dtype_pair, idx_dtype_pair) \
  REGISTER_USER_KERNEL("embedding_prefetch")                                                    \
      .SetCreateFn<EmbeddingPrefetchKernel<OF_PP_PAIR_FIRST(t_dtype_pair),                      \
                                           OF_PP_PAIR_FIRST(table_dtype_pair),                  \
                                           OF_PP_PAIR_FIRST(idx_dtype_pair)>>()                 \
      .SetIsMatchedHob(                                                                         \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                       \
          && (user_op::HobDataType("table_ids", 0) == OF_PP_PAIR_SECOND(table_dtype_pair))      \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair)))  \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                       \
        const user_op::TensorDesc& unique_ids = ctx->InputTensorDesc("unique_ids", 0);          \
        EmbeddingTmpBufferManager buffer_manager(                                               \
            nullptr, unique_ids.shape().elem_cnt(),                                             \
            ctx->Attr<int64_t>("line_size") * sizeof(OF_PP_PAIR_FIRST(t_dtype_pair)), true);    \
        return buffer_manager.TotalBufferSize();                                                \
      });

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_EMBEDDING_PREFETCH_KERNEL, EMBEDDING_DATA_TYPE_SEQ,
                                 TABLE_ID_DATA_TYPE_SEQ, IDX_DATA_TYPE_SEQ)

template<typename T, typename U, typename IDX>
class EmbeddingLookupKernel final : public user_op::OpKernel {
 public:
  EmbeddingLookupKernel() = default;
  ~EmbeddingLookupKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingKernelState<IDX>>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* embedding_state = dynamic_cast<EmbeddingKernelState<IDX>*>(state);
    CHECK(embedding_state != nullptr);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* table_ids = ctx->Tensor4ArgNameAndIndex("table_ids", 0);
    user_op::Tensor* unique_values = ctx->Tensor4ArgNameAndIndex("unique_values", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    uint32_t num_unique;
    LookupAndInitMissing<T, U, IDX>(
        ctx->stream(), embedding_state, unique_ids->shape_view().elem_cnt(), embedding_size,
        line_size, num_unique_ids->dptr(), unique_ids->dptr(), table_ids->dptr(),
        unique_values->mut_dptr<T>(), tmp_buffer->mut_dptr(), &num_unique, false);
    if (ctx->has_output("embeddings", 0)) {
      user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
      CopyValuesToEmbeddings<T>(ctx->stream(), num_unique, embedding_size, line_size,
                                unique_values->data_type(), embeddings->data_type(),
                                unique_values->dptr<T>(), embeddings->mut_dptr());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL(t_dtype_pair, table_dtype_pair, idx_dtype_pair)  \
  REGISTER_USER_KERNEL("embedding_lookup")                                                     \
      .SetCreateFn<EmbeddingLookupKernel<OF_PP_PAIR_FIRST(t_dtype_pair),                       \
                                         OF_PP_PAIR_FIRST(table_dtype_pair),                   \
                                         OF_PP_PAIR_FIRST(idx_dtype_pair)>>()                  \
      .SetIsMatchedHob(                                                                        \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                      \
          && (user_op::HobDataType("unique_values", 0) == OF_PP_PAIR_SECOND(t_dtype_pair))     \
          && (user_op::HobDataType("table_ids", 0) == OF_PP_PAIR_SECOND(table_dtype_pair))     \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))) \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                      \
        const user_op::TensorDesc& unique_ids = ctx->InputTensorDesc("unique_ids", 0);         \
        EmbeddingTmpBufferManager buffer_manager(                                              \
            nullptr, unique_ids.shape().elem_cnt(),                                            \
            ctx->Attr<int64_t>("line_size") * sizeof(OF_PP_PAIR_FIRST(t_dtype_pair)), false);  \
        return buffer_manager.TotalBufferSize();                                               \
      });

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL, EMBEDDING_DATA_TYPE_SEQ,
                                 TABLE_ID_DATA_TYPE_SEQ, IDX_DATA_TYPE_SEQ)

template<typename IDX>
class EmbeddingPutKernel final : public user_op::OpKernel {
 public:
  EmbeddingPutKernel() = default;
  ~EmbeddingPutKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingPutKernelState<IDX>>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* embedding_state = dynamic_cast<EmbeddingPutKernelState<IDX>*>(state);
    CHECK(embedding_state != nullptr);
    embedding::KeyValueStore* store = embedding_state->KeyValueStore();
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* unique_embeddings = ctx->Tensor4ArgNameAndIndex("unique_embeddings", 0);

    IDX* host_num_keys = reinterpret_cast<IDX*>(embedding_state->HostNumKeys());
    OF_CUDA_CHECK(hipMemcpyAsync(host_num_keys, num_unique_ids->dptr(), sizeof(IDX),
                                  hipMemcpyDefault,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));
    CHECK_JUST(ctx->stream()->Sync());

    store->Put(ctx->stream(), *host_num_keys, unique_ids->dptr(), unique_embeddings->dptr());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_EMBEDDING_PUT_KERNEL(dtype, typeproto)           \
  REGISTER_USER_KERNEL("embedding_put")                                \
      .SetCreateFn<EmbeddingPutKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("num_unique_ids", 0) == typeproto));

OF_PP_FOR_EACH_TUPLE(REGISTER_CUDA_EMBEDDING_PUT_KERNEL, IDX_DATA_TYPE_SEQ)

}  // namespace oneflow
