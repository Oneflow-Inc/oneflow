#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/embedding/key_value_store.h"
#include "oneflow/core/embedding/embedding_manager.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/user/kernels/random_mask_generator.h"
#include "oneflow/core/framework/random_generator_impl.h"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/core/ep/include/primitive/copy_nd.h"
#include "oneflow/core/ep/include/primitive/cast.h"
#include "oneflow/core/ep/include/device.h"

namespace oneflow {

namespace {

enum class InitializerType { kUniform, kNormal, kConstant };

struct EmbeddingInitializer {
  InitializerType type;
  union {
    struct {
      float low;
      float high;
    } uniform_param;
    struct {
      float mean;
      float std;
    } normal_param;
    struct {
      float value;
    } constant_param;
  };

  bool operator==(const EmbeddingInitializer& rhs) const {
    if (this->type != rhs.type) { return false; }
    if (rhs.type == InitializerType::kUniform) {
      return (this->uniform_param.low == rhs.uniform_param.low)
             && (this->uniform_param.high == rhs.uniform_param.high);
    } else if (rhs.type == InitializerType::kNormal) {
      return (this->normal_param.mean == rhs.normal_param.mean)
             && (this->normal_param.std == rhs.normal_param.std);
    } else if (rhs.type == InitializerType::kConstant) {
      return this->constant_param.value == rhs.constant_param.value;
    } else {
      UNIMPLEMENTED();
      return false;
    }
  }
};

void ParseInitializerFromJson(const nlohmann::json& initializer,
                              EmbeddingInitializer* embedding_initializer) {
  CHECK(initializer.contains("type"));
  CHECK(initializer["type"].is_string());
  std::string type = initializer["type"].get<std::string>();
  if (type == "uniform") {
    embedding_initializer->type = InitializerType::kUniform;
    CHECK(initializer.contains("low"));
    CHECK(initializer.contains("high"));
    CHECK(initializer["low"].is_number());
    CHECK(initializer["high"].is_number());
    embedding_initializer->uniform_param.low = initializer["low"];
    embedding_initializer->uniform_param.high = initializer["high"];
  } else if (type == "normal") {
    CHECK(initializer.contains("mean"));
    CHECK(initializer.contains("std"));
    CHECK(initializer["mean"].is_number());
    CHECK(initializer["std"].is_number());
    embedding_initializer->type = InitializerType::kNormal;
    embedding_initializer->normal_param.mean = initializer["mean"];
    embedding_initializer->normal_param.std = initializer["std"];
  } else if (type == "constant") {
    CHECK(initializer.contains("value"));
    CHECK(initializer["value"].is_number());
    embedding_initializer->type = InitializerType::kConstant;
    embedding_initializer->constant_param.value = initializer["value"];
  } else {
    UNIMPLEMENTED() << "Unsupported initializer type";
  }
}

int32_t ParseJsonToUniqueInitializerVecAndReturnOffset(
    const nlohmann::json& initializer, std::vector<EmbeddingInitializer>* initializers) {
  EmbeddingInitializer embedding_initializer;
  ParseInitializerFromJson(initializer, &embedding_initializer);
  for (int32_t i = 0; i < initializers->size(); ++i) {
    if (initializers->at(i) == embedding_initializer) { return i; }
  }
  initializers->push_back(embedding_initializer);
  return initializers->size() - 1;
}

void SetInitializerIndex(int32_t row_id, int32_t col_start, int32_t col_end, int64_t line_size,
                         int8_t index, std::vector<int8_t>* initializer_index) {
  int64_t row_offset = row_id * line_size;
  for (int32_t col = col_start; col < col_end; ++col) {
    initializer_index->at(row_offset + col) = index;
  }
}

void ParseAndSetStateInitializerIndex(const std::string& state_initializer,
                                      const int32_t num_tables, const int64_t line_size,
                                      const int64_t embedding_size,
                                      std::vector<EmbeddingInitializer>* initializer_params,
                                      std::vector<int8_t>* initializer_index) {
  if (line_size == embedding_size) { return; }
  CHECK(!state_initializer.empty());
  auto initializers = nlohmann::json::parse(state_initializer);
  CHECK(initializers.is_array());
  const int num_states = line_size / embedding_size - 1;
  CHECK_EQ(num_states, initializers.size());
  for (int32_t i = 0; i < num_states; ++i) {
    int32_t offset =
        ParseJsonToUniqueInitializerVecAndReturnOffset(initializers.at(i), initializer_params);
    int32_t col_start = embedding_size + i * embedding_size;
    int32_t col_end = col_start + embedding_size;
    CHECK_LE(col_end, line_size);
    for (int32_t j = 0; j < num_tables; ++j) {
      SetInitializerIndex(j, col_start, col_end, line_size, offset, initializer_index);
    }
  }
}

void ParseAndSetModelInitializerIndex(const nlohmann::json& tables,
                                      const std::vector<int64_t>& column_dims,
                                      const int32_t num_tables, const int32_t num_columns,
                                      const int64_t line_size, const int64_t embedding_size,
                                      std::vector<EmbeddingInitializer>* initializer_params,
                                      std::vector<int8_t>* initializer_index) {
  for (int32_t i = 0; i < num_tables; ++i) {
    auto table = tables.at(i);
    CHECK(table.contains("columns"));
    auto columns = table["columns"];
    CHECK(columns.is_array());
    CHECK_EQ(num_columns, columns.size()) << "columns size must equal to num embedding dims";
    int32_t col_start = 0;
    for (int k = 0; k < columns.size(); ++k) {
      auto column = columns.at(k);
      CHECK(column.contains("initializer"));
      int32_t offset =
          ParseJsonToUniqueInitializerVecAndReturnOffset(column["initializer"], initializer_params);
      int32_t col_end = col_start + column_dims.at(k);
      SetInitializerIndex(i, col_start, col_end, line_size, offset, initializer_index);
      col_start = col_end;
    }
    CHECK_EQ(col_start, embedding_size);
  }
}

void ParseInitializers(const int64_t line_size, const int64_t embedding_size,
                       const std::string& state_initializer, const std::string& json_serialized,
                       std::vector<EmbeddingInitializer>* initializer_params,
                       std::vector<int8_t>* initializer_index) {
  auto json_object = nlohmann::json::parse(json_serialized);
  CHECK(json_object.contains("column_dims"));
  std::vector<int64_t> column_dims = json_object["column_dims"];
  const int32_t num_columns = column_dims.size();
  CHECK(json_object.contains("tables"));
  auto tables = json_object["tables"];
  CHECK(tables.is_array());
  const int32_t num_tables = tables.size();
  initializer_index->resize(num_tables * line_size);
  ParseAndSetStateInitializerIndex(state_initializer, num_tables, line_size, embedding_size,
                                   initializer_params, initializer_index);
  ParseAndSetModelInitializerIndex(tables, column_dims, num_tables, num_columns, line_size,
                                   embedding_size, initializer_params, initializer_index);
}

template<typename IDX>
class EmbeddingKernelState final : public user_op::OpKernelState {
 public:
  explicit EmbeddingKernelState(user_op::KernelInitContext* ctx)
      : device_index_(-1), generator_(CHECK_JUST(one::MakeGenerator(DeviceType::kCUDA))) {
    OF_CUDA_CHECK(hipGetDevice(&device_index_));
    OF_CUDA_CHECK(hipHostMalloc(&host_num_keys_, sizeof(IDX)));
    const std::string& embedding_name = ctx->Attr<std::string>("embedding_name");
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    key_value_store_ = Singleton<embedding::EmbeddingManager>::Get()->GetKeyValueStore(
        embedding_name, parallel_id);
    uint32_t max_query_length =
        ctx->TensorDesc4ArgNameAndIndex("unique_ids", 0)->shape().elem_cnt();
    key_value_store_->ReserveQueryLength(max_query_length);
    embedding_state_ = Singleton<embedding::EmbeddingManager>::Get()->GetEmbeddingState(
        embedding_name, parallel_id);

    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const std::string& state_initializer = ctx->Attr<std::string>("state_initializer");

    std::vector<EmbeddingInitializer> initializer_param;
    std::vector<int8_t> initializer_index;
    ParseInitializers(line_size, embedding_size, state_initializer,
                      ctx->Attr<std::string>("embedding_tables"), &initializer_param,
                      &initializer_index);

    const size_t param_size_bytes = initializer_param.size() * sizeof(EmbeddingInitializer);
    OF_CUDA_CHECK(hipHostMalloc(&host_initializer_param_, param_size_bytes));
    std::memcpy(host_initializer_param_, initializer_param.data(), param_size_bytes);
    OF_CUDA_CHECK(hipMalloc(&device_initializer_param_, param_size_bytes));
    OF_CUDA_CHECK(hipMemcpyAsync(device_initializer_param_, host_initializer_param_,
                                  param_size_bytes, hipMemcpyDefault,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));

    const size_t index_size_bytes = initializer_index.size() * sizeof(int8_t);
    OF_CUDA_CHECK(hipHostMalloc(&host_initializer_index_, index_size_bytes));
    std::memcpy(host_initializer_index_, initializer_index.data(), index_size_bytes);
    OF_CUDA_CHECK(hipMalloc(&device_initializer_index_, index_size_bytes));
    OF_CUDA_CHECK(hipMemcpyAsync(device_initializer_index_, host_initializer_index_,
                                  index_size_bytes, hipMemcpyDefault,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));
  }
  ~EmbeddingKernelState() override {
    CudaCurrentDeviceGuard guard(device_index_);
    OF_CUDA_CHECK(hipHostFree(host_num_keys_));
    OF_CUDA_CHECK(hipHostFree(host_initializer_param_));
    OF_CUDA_CHECK(hipFree(device_initializer_param_));
    OF_CUDA_CHECK(hipHostFree(host_initializer_index_));
    OF_CUDA_CHECK(hipFree(device_initializer_index_));
  }

  void* HostNumKeys() { return host_num_keys_; }

  embedding::KeyValueStore* KeyValueStore() { return key_value_store_; }

  embedding::EmbeddingState* EmbeddingState() { return embedding_state_; }

  one::Generator* generator() { return generator_.get(); }

  const int8_t* InitializerIndex() { return device_initializer_index_; }
  const EmbeddingInitializer* Initializers() { return device_initializer_param_; }

 private:
  int device_index_;
  void* host_num_keys_;
  std::shared_ptr<one::Generator> generator_;
  embedding::KeyValueStore* key_value_store_;
  embedding::EmbeddingState* embedding_state_;

  EmbeddingInitializer* host_initializer_param_;
  EmbeddingInitializer* device_initializer_param_;
  int8_t* host_initializer_index_;
  int8_t* device_initializer_index_;
};

class EmbeddingPutKernelState final : public user_op::OpKernelState {
 public:
  explicit EmbeddingPutKernelState(user_op::KernelInitContext* ctx) {
    const std::string& embedding_name = ctx->Attr<std::string>("embedding_name");
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    key_value_store_ = Singleton<embedding::EmbeddingManager>::Get()->GetKeyValueStore(
        embedding_name, parallel_id);
    uint32_t max_query_length =
        ctx->TensorDesc4ArgNameAndIndex("unique_ids", 0)->shape().elem_cnt();
    key_value_store_->ReserveQueryLength(max_query_length);
    embedding_state_ = Singleton<embedding::EmbeddingManager>::Get()->GetEmbeddingState(
        embedding_name, parallel_id);
  }
  ~EmbeddingPutKernelState() override = default;

  embedding::KeyValueStore* KeyValueStore() { return key_value_store_; }
  embedding::EmbeddingState* EmbeddingState() { return embedding_state_; }

 private:
  embedding::KeyValueStore* key_value_store_;
  embedding::EmbeddingState* embedding_state_;
};

template<typename T, typename U>
__global__ void InitValueKernel(uint64_t seed, one::CUDAGeneratorState* cuda_gen_state,
                                uint64_t inc_offset, const int32_t line_size,
                                const int32_t embedding_size,
                                const EmbeddingInitializer* initializer_param,
                                const int8_t* initializer_index, const U* table_ids,
                                const uint32_t* num_missing_keys, const uint32_t* missing_indices,
                                T* values) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, cuda_gen_state->dev_offset, &state);
  int64_t n = *num_missing_keys * line_size;
  CUDA_1D_KERNEL_LOOP(i, n) {
    int row = i / line_size;
    int col = i - row * line_size;
    const uint32_t index = missing_indices[row];
    const int64_t offset = index * line_size + col;
    const int32_t table_idx = table_ids[index];
    const int32_t initializer_idx = initializer_index[table_idx * line_size + col];
    EmbeddingInitializer initializer = initializer_param[initializer_idx];
    T value;
    if (initializer.type == InitializerType::kUniform) {
      const float low = initializer.uniform_param.low;
      const float high = initializer.uniform_param.high;
      value = hiprand_uniform(&state) * (high - low) + low;
    } else if (initializer.type == InitializerType::kNormal) {
      const float mean = initializer.normal_param.mean;
      const float std = initializer.normal_param.std;
      value = hiprand_normal(&state) * std + mean;
    } else if (initializer.type == InitializerType::kConstant) {
      value = initializer.constant_param.value;
    } else {
      __trap();
    }
    values[offset] = value;
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    int32_t new_counter = cuda::atomic::Add(&cuda_gen_state->dev_counter, 1) + 1;
    if (new_counter == gridDim.x) {
      cuda_gen_state->dev_counter = 0;           // reset counter to zero
      cuda_gen_state->dev_offset += inc_offset;  // maintain the state of generator's dev_offset
    }
  }
}

template<typename T, typename U, typename IDX>
void LookupAndInitMissing(ep::Stream* stream, EmbeddingKernelState<IDX>* kernel_state,
                          uint32_t num_unique, const int64_t embedding_size,
                          const int64_t line_size, const bool is_prefetch, const void* unique_ids,
                          const void* table_ids, void* num_missing_ptr, void* missing_indices,
                          void* store_values) {
  const auto& generator = kernel_state->generator();
  CHECK_NOTNULL(generator);
  std::shared_ptr<one::CUDAGeneratorImpl> cuda_generator =
      CHECK_JUST(generator->template Get<one::CUDAGeneratorImpl>(stream->device()->device_index()));
  uint64_t seed = cuda_generator->current_seed();
  one::CUDAGeneratorState* cuda_gen_state = cuda_generator->cuda_gen_state();
  embedding::KeyValueStore* store = kernel_state->KeyValueStore();
  const EmbeddingInitializer* initializer_param = kernel_state->Initializers();
  const int8_t* initializer_index = kernel_state->InitializerIndex();
  store->Get(stream, num_unique, unique_ids, store_values,
             reinterpret_cast<uint32_t*>(num_missing_ptr),
             reinterpret_cast<uint32_t*>(missing_indices));
  void* host_num_keys = kernel_state->HostNumKeys();
  CHECK_GE(sizeof(IDX), sizeof(uint32_t));  // host_num_keys's buffer size is sizeof(IDX)
  OF_CUDA_CHECK(hipMemcpyAsync(host_num_keys, num_missing_ptr, sizeof(uint32_t), hipMemcpyDefault,
                                stream->As<ep::CudaStream>()->hip_stream()));
  CHECK_JUST(stream->Sync());
  uint32_t num_missing = *reinterpret_cast<uint32_t*>(host_num_keys);
  // init missing values
  if (num_missing > 0) {
    const int64_t elem_cnt = num_missing * line_size;
    const int64_t num_blocks = BlocksNum4ThreadsNum(elem_cnt);
    const uint64_t inc_offset = std::ceil(elem_cnt / num_blocks / kCudaThreadsNumPerBlock);
    InitValueKernel<T, U>
        <<<num_blocks, kCudaThreadsNumPerBlock, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            seed, cuda_gen_state, inc_offset, line_size, embedding_size, initializer_param,
            initializer_index, reinterpret_cast<const U*>(table_ids),
            reinterpret_cast<uint32_t*>(num_missing_ptr),
            reinterpret_cast<uint32_t*>(missing_indices), reinterpret_cast<T*>(store_values));
  }
  if (is_prefetch) { store->Put(stream, num_unique, unique_ids, store_values); }
}

template<typename T, size_t pack_size>
struct alignas(sizeof(T) * pack_size) Pack {
  T elem[pack_size];
};

template<typename T, typename U, typename V, int pack_size>
__global__ void FusedInitSliceCast(const int32_t elem_cnt, uint64_t seed,
                                   one::CUDAGeneratorState* cuda_gen_state, uint64_t inc_offset,
                                   const int32_t line_size, const int32_t embedding_size,
                                   const int32_t line_num_pack, const int32_t embedding_num_pack,
                                   const EmbeddingInitializer* initializer_param,
                                   const int8_t* initializer_index, const U* table_ids,
                                   const uint8_t* lookup_mask, Pack<T, pack_size>* values,
                                   Pack<V, pack_size>* embeddings) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, cuda_gen_state->dev_offset, &state);
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    int row = i / line_num_pack;
    int col = i - row * line_num_pack;
    Pack<T, pack_size> value_i;
    if (!lookup_mask[row]) {
      const int32_t table_idx = table_ids[row];
#pragma unroll
      for (int k = 0; k < pack_size; ++k) {
        const int32_t initializer_idx =
            initializer_index[table_idx * line_size + col * pack_size + k];
        EmbeddingInitializer initializer = initializer_param[initializer_idx];
        T value;
        if (initializer.type == InitializerType::kUniform) {
          const float low = initializer.uniform_param.low;
          const float high = initializer.uniform_param.high;
          value = hiprand_uniform(&state) * (high - low) + low;
        } else if (initializer.type == InitializerType::kNormal) {
          const float mean = initializer.normal_param.mean;
          const float std = initializer.normal_param.std;
          value = hiprand_normal(&state) * std + mean;
        } else if (initializer.type == InitializerType::kConstant) {
          value = initializer.constant_param.value;
        } else {
          __trap();
        }
        value_i.elem[k] = value;
      }
      values[i] = value_i;
    } else {
      value_i = values[i];
    }
    if (embeddings != nullptr && col < embedding_num_pack) {
      int64_t embedding_offset = row * embedding_num_pack + col;
      Pack<V, pack_size> embedding_i;
#pragma unroll
      for (int k = 0; k < pack_size; ++k) { embedding_i.elem[k] = static_cast<V>(value_i.elem[k]); }
      embeddings[embedding_offset] = embedding_i;
    }
  }
}

template<typename T, typename U, typename V>
void InitMissingAndSliceCast(hipStream_t hip_stream, uint32_t num_unique,
                             const int64_t embedding_size, const int64_t line_size, uint64_t seed,
                             one::CUDAGeneratorState* cuda_gen_state,
                             const EmbeddingInitializer* initializer_param,
                             const int8_t* initializer_index, const void* table_ids,
                             const uint8_t* mask, T* values_ptr, V* embeddings_ptr) {
  int32_t pack_size;
  if (embedding_size % 4 == 0 && line_size % 4 == 0) {
    pack_size = 4;
  } else if (embedding_size % 2 == 0 && line_size % 2 == 0) {
    pack_size = 2;
  } else {
    pack_size = 1;
  }
  int32_t embedding_num_pack = embedding_size / pack_size;
  int32_t line_num_pack = line_size / pack_size;
  int64_t value_elem_cnt = num_unique * line_size;
  int64_t value_elem_num_pack = value_elem_cnt / pack_size;
  const int64_t num_blocks = BlocksNum4ThreadsNum(value_elem_num_pack);
  const uint64_t inc_offset = std::ceil(value_elem_cnt / num_blocks / kCudaThreadsNumPerBlock);
  if (pack_size == 4) {
    FusedInitSliceCast<T, U, V, 4><<<num_blocks, kCudaThreadsNumPerBlock, 0, hip_stream>>>(
        value_elem_num_pack, seed, cuda_gen_state, inc_offset, line_size, embedding_size,
        line_num_pack, embedding_num_pack, initializer_param, initializer_index,
        reinterpret_cast<const U*>(table_ids), mask, reinterpret_cast<Pack<T, 4>*>(values_ptr),
        reinterpret_cast<Pack<V, 4>*>(embeddings_ptr));
  } else if (pack_size == 2) {
    FusedInitSliceCast<T, U, V, 2><<<num_blocks, kCudaThreadsNumPerBlock, 0, hip_stream>>>(
        value_elem_num_pack, seed, cuda_gen_state, inc_offset, line_size, embedding_size,
        line_num_pack, embedding_num_pack, initializer_param, initializer_index,
        reinterpret_cast<const U*>(table_ids), mask, reinterpret_cast<Pack<T, 2>*>(values_ptr),
        reinterpret_cast<Pack<V, 2>*>(embeddings_ptr));
  } else {
    FusedInitSliceCast<T, U, V, 1><<<num_blocks, kCudaThreadsNumPerBlock, 0, hip_stream>>>(
        value_elem_num_pack, seed, cuda_gen_state, inc_offset, line_size, embedding_size,
        line_num_pack, embedding_num_pack, initializer_param, initializer_index,
        reinterpret_cast<const U*>(table_ids), mask, reinterpret_cast<Pack<T, 1>*>(values_ptr),
        reinterpret_cast<Pack<V, 1>*>(embeddings_ptr));
  }
}

template<typename T, typename U, typename IDX>
void LookupAndFusedInitMissingSliceCast(ep::Stream* stream, EmbeddingKernelState<IDX>* kernel_state,
                                        uint32_t num_unique, const int64_t embedding_size,
                                        const int64_t line_size, DataType value_dtype,
                                        DataType embedding_dtype, const void* unique_ids,
                                        const void* table_ids, uint8_t* lookup_mask_ptr,
                                        void* values_ptr, void* embeddings_ptr) {
  const auto& generator = kernel_state->generator();
  CHECK_NOTNULL(generator);
  std::shared_ptr<one::CUDAGeneratorImpl> cuda_generator =
      CHECK_JUST(generator->template Get<one::CUDAGeneratorImpl>(stream->device()->device_index()));
  uint64_t seed = cuda_generator->current_seed();
  one::CUDAGeneratorState* cuda_gen_state = cuda_generator->cuda_gen_state();
  embedding::KeyValueStore* store = kernel_state->KeyValueStore();
  const EmbeddingInitializer* initializer_param = kernel_state->Initializers();
  const int8_t* initializer_index = kernel_state->InitializerIndex();
  hipStream_t hip_stream = stream->As<ep::CudaStream>()->hip_stream();
  store->Get(stream, num_unique, unique_ids, values_ptr, lookup_mask_ptr);
  if (embedding_dtype == value_dtype) {
    InitMissingAndSliceCast<T, U, T>(
        hip_stream, num_unique, embedding_size, line_size, seed, cuda_gen_state, initializer_param,
        initializer_index, reinterpret_cast<const U*>(table_ids), lookup_mask_ptr,
        reinterpret_cast<T*>(values_ptr), reinterpret_cast<T*>(embeddings_ptr));
  } else if (embedding_dtype == DataType::kFloat16) {
    InitMissingAndSliceCast<T, U, half>(
        hip_stream, num_unique, embedding_size, line_size, seed, cuda_gen_state, initializer_param,
        initializer_index, reinterpret_cast<const U*>(table_ids), lookup_mask_ptr,
        reinterpret_cast<T*>(values_ptr), reinterpret_cast<half*>(embeddings_ptr));
  } else {
    UNIMPLEMENTED() << "Unimplemented data_type " << embedding_dtype;
  }
}

template<typename T, typename U>
__global__ void Copy2D(int64_t out_elem_cnt, const int32_t in_cols, const int32_t out_cols,
                       const T* in, U* out) {
  CUDA_1D_KERNEL_LOOP(i, out_elem_cnt) {
    const int32_t row = i / out_cols;
    const int32_t col = i - row * out_cols;
    const int64_t in_offset = row * in_cols + col;
    out[i] = static_cast<U>(in[in_offset]);
  }
}

template<typename T>
void CopyValuesToEmbeddings(ep::Stream* stream, int64_t num_unique, const int32_t embedding_size,
                            const int32_t value_size, const DataType value_dtype,
                            const DataType embedding_dtype, const T* values, void* embeddings) {
  bool need_cast = (value_dtype != embedding_dtype);
  bool need_copy_nd = (embedding_size != value_size);
  CHECK(need_cast || need_copy_nd);
  if (need_cast && !need_copy_nd) {
    const int64_t cast_elem_count = num_unique * embedding_size;
    std::unique_ptr<ep::primitive::Cast> cast_primitive =
        ep::primitive::NewPrimitive<ep::primitive::CastFactory>(DeviceType::kCUDA, value_dtype,
                                                                embedding_dtype);
    cast_primitive->Launch(stream, values, embeddings, cast_elem_count);
  } else if (!need_cast && need_copy_nd) {
    const int32_t ndims = 2;
    DimVector src_pos_vec(ndims, 0);
    DimVector dst_pos_vec(ndims, 0);
    DimVector src_shape = {num_unique, value_size};
    DimVector dst_shape = {num_unique, embedding_size};
    DimVector extent_shape = {num_unique, embedding_size};
    std::unique_ptr<ep::primitive::CopyNd> copy_nd_primitive =
        ep::primitive::NewPrimitive<ep::primitive::CopyNdFactory>(DeviceType::kCUDA, ndims);
    CHECK(copy_nd_primitive);
    copy_nd_primitive->Launch(stream, value_dtype, ndims, embeddings, dst_shape.data(),
                              dst_pos_vec.data(), values, src_shape.data(), src_pos_vec.data(),
                              extent_shape.data());
  } else {
    const int64_t embedding_elem_cnt = num_unique * embedding_size;
    if (embedding_dtype == DataType::kFloat16) {
      Copy2D<T, half><<<BlocksNum4ThreadsNum(embedding_elem_cnt), kCudaThreadsNumPerBlock, 0,
                        stream->As<ep::CudaStream>()->hip_stream()>>>(
          embedding_elem_cnt, value_size, embedding_size, values,
          reinterpret_cast<half*>(embeddings));
    } else {
      UNIMPLEMENTED();
    }
  }
}

template<typename T, bool is_prefetch>
user_op::InferTmpSizeFn GenEmbeddingInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    size_t total_buffer_size = 0;
    if (embedding::UseDynamicMemoryAllocation()) { return total_buffer_size; }
    const user_op::TensorDesc& unique_ids = ctx->InputTensorDesc("unique_ids", 0);
    int64_t num_ids = unique_ids.shape().elem_cnt();
    size_t num_missing_size = GetCudaAlignedSize(sizeof(uint32_t));
    size_t missing_indices_size = GetCudaAlignedSize(num_ids * sizeof(uint32_t));
    size_t value_buffer_size;
    if (is_prefetch) {
      size_t value_byte_size = ctx->Attr<int64_t>("line_size") * sizeof(T);
      value_buffer_size = num_ids * value_byte_size;
    } else {
      value_buffer_size = 0;
    }
    total_buffer_size = num_missing_size + missing_indices_size + value_buffer_size;
    return total_buffer_size;
  };
}

}  // namespace

template<typename T, typename U, typename IDX>
class EmbeddingPrefetchKernel final : public user_op::OpKernel {
 public:
  EmbeddingPrefetchKernel() : current_iter_(0){};
  ~EmbeddingPrefetchKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingKernelState<IDX>>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingKernelState<IDX>*>(state);
    CHECK(kernel_state != nullptr);
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingPrefetchStart(ctx, current_iter_);
    uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* table_ids = ctx->Tensor4ArgNameAndIndex("table_ids", 0);
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");

    void* num_missing_ptr;
    embedding_state->AllocPrefetchTmpBuffer(ctx, &num_missing_ptr,
                                            GetCudaAlignedSize(sizeof(uint32_t)));
    void* missing_indices_ptr;
    embedding_state->AllocPrefetchTmpBuffer(ctx, &missing_indices_ptr,
                                            GetCudaAlignedSize(num_unique * sizeof(uint32_t)));
    void* values_ptr;
    embedding_state->AllocPrefetchTmpBuffer(ctx, &values_ptr,
                                            GetCudaAlignedSize(num_unique * line_size * sizeof(T)));
    LookupAndInitMissing<T, U, IDX>(ctx->stream(), kernel_state, num_unique, embedding_size,
                                    line_size, true, unique_ids->dptr(), table_ids->dptr(),
                                    num_missing_ptr, missing_indices_ptr, values_ptr);
    embedding_state->FreePrefetchTmpBuffer(ctx, num_missing_ptr);
    embedding_state->FreePrefetchTmpBuffer(ctx, missing_indices_ptr);
    embedding_state->FreePrefetchTmpBuffer(ctx, values_ptr);
    embedding_state->OnEmbeddingPrefetchEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define EMBEDDING_DATA_TYPE_SEQ OF_PP_MAKE_TUPLE_SEQ(float, DataType::kFloat)

#define TABLE_ID_DATA_TYPE_SEQ                      \
  OF_PP_MAKE_TUPLE_SEQ(uint8_t, DataType::kUInt8)   \
  OF_PP_MAKE_TUPLE_SEQ(uint32_t, DataType::kUInt32) \
  OF_PP_MAKE_TUPLE_SEQ(uint64_t, DataType::kUInt64) \
  OF_PP_MAKE_TUPLE_SEQ(int8_t, DataType::kInt8)     \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32)   \
  OF_PP_MAKE_TUPLE_SEQ(int64_t, DataType::kInt64)

#define IDX_DATA_TYPE_SEQ                           \
  OF_PP_MAKE_TUPLE_SEQ(uint32_t, DataType::kUInt32) \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32)

#define REGISTER_CUDA_EMBEDDING_PREFETCH_KERNEL(t_dtype_pair, table_dtype_pair, idx_dtype_pair) \
  REGISTER_USER_KERNEL("embedding_prefetch")                                                    \
      .SetCreateFn<EmbeddingPrefetchKernel<OF_PP_PAIR_FIRST(t_dtype_pair),                      \
                                           OF_PP_PAIR_FIRST(table_dtype_pair),                  \
                                           OF_PP_PAIR_FIRST(idx_dtype_pair)>>()                 \
      .SetIsMatchedHob(                                                                         \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                       \
          && (user_op::HobDataType("table_ids", 0) == OF_PP_PAIR_SECOND(table_dtype_pair))      \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair)))  \
      .SetInferTmpSizeFn(GenEmbeddingInferTmpSizeFn<OF_PP_PAIR_FIRST(t_dtype_pair), true>());

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_EMBEDDING_PREFETCH_KERNEL, EMBEDDING_DATA_TYPE_SEQ,
                                 TABLE_ID_DATA_TYPE_SEQ, IDX_DATA_TYPE_SEQ)

template<typename T, typename U, typename IDX>
class EmbeddingLookupKernel final : public user_op::OpKernel {
 public:
  EmbeddingLookupKernel() : current_iter_(0){};
  ~EmbeddingLookupKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingKernelState<IDX>>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingKernelState<IDX>*>(state);
    CHECK(kernel_state != nullptr);
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingLookupStart(ctx, current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* table_ids = ctx->Tensor4ArgNameAndIndex("table_ids", 0);
    user_op::Tensor* unique_values = ctx->Tensor4ArgNameAndIndex("unique_values", 0);
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const bool has_output_embeddings = ctx->has_output("embeddings", 0);
    uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    void* values_ptr = embedding_state->LookupUniqueValues(current_iter_);
    if (has_output_embeddings && kernel_state->KeyValueStore()->IsFusionSupported()) {
      void* embeddings_ptr = embedding_state->LookupEmbeddings(current_iter_);
      user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
      void* lookup_mask_ptr;
      embedding_state->AllocTmpBuffer(ctx, &lookup_mask_ptr,
                                      GetCudaAlignedSize(num_unique * sizeof(uint8_t)));
      LookupAndFusedInitMissingSliceCast<T, U, IDX>(
          ctx->stream(), kernel_state, num_unique, embedding_size, line_size,
          unique_values->data_type(), embeddings->data_type(), unique_ids->dptr(),
          table_ids->dptr(), reinterpret_cast<uint8_t*>(lookup_mask_ptr), values_ptr,
          embeddings_ptr);
      embedding_state->FreeTmpBuffer(ctx, lookup_mask_ptr);
    } else {
      void* num_missing_ptr;
      embedding_state->AllocTmpBuffer(ctx, &num_missing_ptr, GetCudaAlignedSize(sizeof(uint32_t)));
      void* missing_indices_ptr;
      embedding_state->AllocTmpBuffer(ctx, &missing_indices_ptr,
                                      GetCudaAlignedSize(num_unique * sizeof(uint32_t)));
      LookupAndInitMissing<T, U, IDX>(ctx->stream(), kernel_state, num_unique, embedding_size,
                                      line_size, false, unique_ids->dptr(), table_ids->dptr(),
                                      num_missing_ptr, missing_indices_ptr, values_ptr);
      embedding_state->FreeTmpBuffer(ctx, num_missing_ptr);
      embedding_state->FreeTmpBuffer(ctx, missing_indices_ptr);
      if (has_output_embeddings) {
        void* embeddings_ptr = embedding_state->LookupEmbeddings(current_iter_);
        user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
        CopyValuesToEmbeddings<T>(ctx->stream(), num_unique, embedding_size, line_size,
                                  unique_values->data_type(), embeddings->data_type(),
                                  reinterpret_cast<T*>(values_ptr), embeddings_ptr);
      }
    }
    embedding_state->OnEmbeddingLookupEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL(t_dtype_pair, table_dtype_pair, idx_dtype_pair)  \
  REGISTER_USER_KERNEL("embedding_lookup")                                                     \
      .SetCreateFn<EmbeddingLookupKernel<OF_PP_PAIR_FIRST(t_dtype_pair),                       \
                                         OF_PP_PAIR_FIRST(table_dtype_pair),                   \
                                         OF_PP_PAIR_FIRST(idx_dtype_pair)>>()                  \
      .SetIsMatchedHob(                                                                        \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                      \
          && (user_op::HobDataType("unique_values", 0) == OF_PP_PAIR_SECOND(t_dtype_pair))     \
          && (user_op::HobDataType("table_ids", 0) == OF_PP_PAIR_SECOND(table_dtype_pair))     \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))) \
      .SetInferTmpSizeFn(GenEmbeddingInferTmpSizeFn<OF_PP_PAIR_FIRST(t_dtype_pair), false>());

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL, EMBEDDING_DATA_TYPE_SEQ,
                                 TABLE_ID_DATA_TYPE_SEQ, IDX_DATA_TYPE_SEQ)

template<typename IDX>
class EmbeddingPutKernel final : public user_op::OpKernel {
 public:
  EmbeddingPutKernel() : current_iter_(0){};
  ~EmbeddingPutKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingPutKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingPutKernelState*>(state);
    CHECK(kernel_state != nullptr);
    embedding::KeyValueStore* store = kernel_state->KeyValueStore();
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingPutStart(ctx, current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* unique_embeddings = ctx->Tensor4ArgNameAndIndex("unique_embeddings", 0);
    uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    store->Put(ctx->stream(), num_unique, unique_ids->dptr(),
               embedding_state->EmbeddingPutUniqueEmbeddings(current_iter_));
    embedding_state->OnEmbeddingPutEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define REGISTER_CUDA_EMBEDDING_PUT_KERNEL(dtype, typeproto)           \
  REGISTER_USER_KERNEL("embedding_put")                                \
      .SetCreateFn<EmbeddingPutKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("num_unique_ids", 0) == typeproto));

OF_PP_FOR_EACH_TUPLE(REGISTER_CUDA_EMBEDDING_PUT_KERNEL, IDX_DATA_TYPE_SEQ)

template<typename IDX>
class FusedSgdEmbeddingUpdatePutKernel final : public user_op::OpKernel {
 public:
  FusedSgdEmbeddingUpdatePutKernel() : current_iter_(0){};
  ~FusedSgdEmbeddingUpdatePutKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingPutKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingPutKernelState*>(state);
    CHECK(kernel_state != nullptr);
    embedding::KeyValueStore* store = kernel_state->KeyValueStore();
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingFusedUpdatePutStart(ctx, current_iter_);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
    const float* learning_rate_ptr = learning_rate->dptr<float>();
    const auto scale = ctx->Attr<double>("scale");
    uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    store->FusedHalfUpdatePut(
        ctx->stream(), num_unique, unique_ids->dptr(),
        embedding_state->EmbeddingFusedUpdatePutUniqueEmbeddings(current_iter_),
        embedding_grad->dptr(), learning_rate_ptr, scale);
    embedding_state->OnEmbeddingFusedUpdatePutEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define REGISTER_CUDA_FUSED_SGD_EMBEDDING_UPDATE_PUT_KERNEL(dtype, typeproto)                \
  REGISTER_USER_KERNEL("fused_sgd_embedding_update_put")                                     \
      .SetCreateFn<FusedSgdEmbeddingUpdatePutKernel<dtype>>()                                \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                       \
                       && (user_op::HobDataType("num_unique_ids", 0) == typeproto)           \
                       && (user_op::HobDataType("unique_embeddings", 0) == DataType::kFloat) \
                       && (user_op::HobDataType("embedding_grad", 0) == DataType::kFloat16));

OF_PP_FOR_EACH_TUPLE(REGISTER_CUDA_FUSED_SGD_EMBEDDING_UPDATE_PUT_KERNEL, IDX_DATA_TYPE_SEQ)

}  // namespace oneflow
