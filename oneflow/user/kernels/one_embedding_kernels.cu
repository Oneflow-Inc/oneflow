#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/embedding/key_value_store.h"
#include "oneflow/core/embedding/embedding_manager.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/ep/include/primitive/copy_nd.h"
#include "oneflow/core/ep/include/primitive/cast.h"
#include "oneflow/core/ep/include/device.h"
#include "oneflow/user/kernels/one_embedding_data_shuffle.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace oneflow {

namespace {

enum class InitializerType { kUniform, kNormal, kConstant, kTruncNormal };

struct EmbeddingInitializer {
  InitializerType type;
  union {
    struct {
      float low;
      float high;
    } uniform_param;
    struct {
      float mean;
      float std;
    } normal_param;
    struct {
      float value;
    } constant_param;
    struct {
      float mean;
      float std;
      float a;
      float b;
    } trunc_normal_param;
  };

  bool operator==(const EmbeddingInitializer& rhs) const {
    if (this->type != rhs.type) { return false; }
    if (rhs.type == InitializerType::kUniform) {
      return (this->uniform_param.low == rhs.uniform_param.low)
             && (this->uniform_param.high == rhs.uniform_param.high);
    } else if (rhs.type == InitializerType::kNormal) {
      return (this->normal_param.mean == rhs.normal_param.mean)
             && (this->normal_param.std == rhs.normal_param.std);
    } else if (rhs.type == InitializerType::kConstant) {
      return this->constant_param.value == rhs.constant_param.value;
    } else if (rhs.type == InitializerType::kTruncNormal) {
      return (this->trunc_normal_param.mean == rhs.trunc_normal_param.mean)
             && (this->trunc_normal_param.std == rhs.trunc_normal_param.std)
             && (this->trunc_normal_param.a == rhs.trunc_normal_param.a)
             && (this->trunc_normal_param.b == rhs.trunc_normal_param.b);
    } else {
      UNIMPLEMENTED();
      return false;
    }
  }
};

void ParseInitializerFromJson(const nlohmann::json& initializer,
                              EmbeddingInitializer* embedding_initializer) {
  CHECK(initializer.contains("type"));
  CHECK(initializer["type"].is_string());
  std::string type = initializer["type"].get<std::string>();
  if (type == "uniform") {
    embedding_initializer->type = InitializerType::kUniform;
    CHECK(initializer.contains("low"));
    CHECK(initializer.contains("high"));
    CHECK(initializer["low"].is_number());
    CHECK(initializer["high"].is_number());
    embedding_initializer->uniform_param.low = initializer["low"];
    embedding_initializer->uniform_param.high = initializer["high"];
  } else if (type == "normal") {
    CHECK(initializer.contains("mean"));
    CHECK(initializer.contains("std"));
    CHECK(initializer["mean"].is_number());
    CHECK(initializer["std"].is_number());
    embedding_initializer->type = InitializerType::kNormal;
    embedding_initializer->normal_param.mean = initializer["mean"];
    embedding_initializer->normal_param.std = initializer["std"];
  } else if (type == "constant") {
    CHECK(initializer.contains("value"));
    CHECK(initializer["value"].is_number());
    embedding_initializer->type = InitializerType::kConstant;
    embedding_initializer->constant_param.value = initializer["value"];
  } else if (type == "trunc_normal") {
    CHECK(initializer.contains("mean"));
    CHECK(initializer.contains("std"));
    CHECK(initializer.contains("a"));
    CHECK(initializer.contains("b"));
    CHECK(initializer["mean"].is_number());
    CHECK(initializer["std"].is_number());
    CHECK(initializer["a"].is_number());
    CHECK(initializer["b"].is_number());
    embedding_initializer->type = InitializerType::kTruncNormal;
    embedding_initializer->trunc_normal_param.mean = initializer["mean"];
    embedding_initializer->trunc_normal_param.std = initializer["std"];
    embedding_initializer->trunc_normal_param.a = initializer["a"];
    embedding_initializer->trunc_normal_param.b = initializer["b"];
  } else {
    UNIMPLEMENTED() << "Unsupported initializer type";
  }
}

int32_t ParseJsonToUniqueInitializerVecAndReturnOffset(
    const nlohmann::json& initializer, std::vector<EmbeddingInitializer>* initializers) {
  EmbeddingInitializer embedding_initializer;
  ParseInitializerFromJson(initializer, &embedding_initializer);
  for (int32_t i = 0; i < initializers->size(); ++i) {
    if (initializers->at(i) == embedding_initializer) { return i; }
  }
  initializers->push_back(embedding_initializer);
  return initializers->size() - 1;
}

void SetInitializerIndex(int32_t row_id, int32_t col_start, int32_t col_end, int64_t line_size,
                         int8_t index, std::vector<int8_t>* initializer_index) {
  int64_t row_offset = row_id * line_size;
  for (int32_t col = col_start; col < col_end; ++col) {
    initializer_index->at(row_offset + col) = index;
  }
}

void ParseAndSetStateInitializerIndex(const std::string& state_initializer,
                                      const int32_t num_tables, const int64_t line_size,
                                      const int64_t embedding_size,
                                      std::vector<EmbeddingInitializer>* initializer_params,
                                      std::vector<int8_t>* initializer_index) {
  if (line_size == embedding_size) { return; }
  CHECK(!state_initializer.empty());
  auto initializers = nlohmann::json::parse(state_initializer);
  CHECK(initializers.is_array());
  const int num_states = line_size / embedding_size - 1;
  CHECK_EQ(num_states, initializers.size());
  for (int32_t i = 0; i < num_states; ++i) {
    int32_t offset =
        ParseJsonToUniqueInitializerVecAndReturnOffset(initializers.at(i), initializer_params);
    int32_t col_start = embedding_size + i * embedding_size;
    int32_t col_end = col_start + embedding_size;
    CHECK_LE(col_end, line_size);
    for (int32_t j = 0; j < num_tables; ++j) {
      SetInitializerIndex(j, col_start, col_end, line_size, offset, initializer_index);
    }
  }
}

void ParseAndSetStepInitializerIndex(const int32_t num_tables, const int64_t line_size,
                                     const int64_t embedding_size,
                                     std::vector<EmbeddingInitializer>* initializer_params,
                                     std::vector<int8_t>* initializer_index) {
  if (line_size % embedding_size == 0) { return; }
  nlohmann::json initializer;
  initializer["type"] = "constant";
  initializer["value"] = 0.0;
  int32_t offset = ParseJsonToUniqueInitializerVecAndReturnOffset(initializer, initializer_params);
  int32_t col_start = line_size / embedding_size * embedding_size;
  int32_t col_end = line_size;
  CHECK_LE(col_end, line_size);
  for (int32_t j = 0; j < num_tables; ++j) {
    SetInitializerIndex(j, col_start, col_end, line_size, offset, initializer_index);
  }
}

void ParseAndSetModelInitializerIndex(const nlohmann::json& tables,
                                      const std::vector<int64_t>& column_dims,
                                      const int32_t num_tables, const int32_t num_columns,
                                      const int64_t line_size, const int64_t embedding_size,
                                      std::vector<EmbeddingInitializer>* initializer_params,
                                      std::vector<int8_t>* initializer_index) {
  for (int32_t i = 0; i < num_tables; ++i) {
    auto table = tables.at(i);
    CHECK(table.contains("columns"));
    auto columns = table["columns"];
    CHECK(columns.is_array());
    CHECK_EQ(num_columns, columns.size()) << "columns size must equal to num embedding dims";
    int32_t col_start = 0;
    for (int k = 0; k < columns.size(); ++k) {
      auto column = columns.at(k);
      CHECK(column.contains("initializer"));
      int32_t offset =
          ParseJsonToUniqueInitializerVecAndReturnOffset(column["initializer"], initializer_params);
      int32_t col_end = col_start + column_dims.at(k);
      SetInitializerIndex(i, col_start, col_end, line_size, offset, initializer_index);
      col_start = col_end;
    }
    CHECK_EQ(col_start, embedding_size);
  }
}

void ParseInitializers(const int64_t line_size, const int64_t embedding_size,
                       const std::string& state_initializer, const std::string& json_serialized,
                       std::vector<EmbeddingInitializer>* initializer_params,
                       std::vector<int8_t>* initializer_index) {
  auto json_object = nlohmann::json::parse(json_serialized);
  CHECK(json_object.contains("column_dims"));
  std::vector<int64_t> column_dims = json_object["column_dims"];
  const int32_t num_columns = column_dims.size();
  CHECK(json_object.contains("tables"));
  auto tables = json_object["tables"];
  CHECK(tables.is_array());
  const int32_t num_tables = tables.size();
  initializer_index->resize(num_tables * line_size);
  ParseAndSetStepInitializerIndex(num_tables, line_size, embedding_size, initializer_params,
                                  initializer_index);
  ParseAndSetStateInitializerIndex(state_initializer, num_tables, line_size, embedding_size,
                                   initializer_params, initializer_index);
  ParseAndSetModelInitializerIndex(tables, column_dims, num_tables, num_columns, line_size,
                                   embedding_size, initializer_params, initializer_index);
}

template<typename IDX>
class EmbeddingKernelState final : public user_op::OpKernelState {
 public:
  explicit EmbeddingKernelState(user_op::KernelInitContext* ctx) : device_index_(-1) {
    OF_CUDA_CHECK(hipGetDevice(&device_index_));
    OF_CUDA_CHECK(hipHostMalloc(&host_num_keys_, sizeof(IDX)));
    const std::string& embedding_name = ctx->Attr<std::string>("embedding_name");
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    key_value_store_ = Singleton<embedding::EmbeddingManager>::Get()->GetKeyValueStore(
        embedding_name, parallel_id);
    uint32_t max_query_length =
        ctx->TensorDesc4ArgNameAndIndex("unique_ids", 0)->shape().elem_cnt();
    key_value_store_->ReserveQueryLength(max_query_length);
    embedding_state_ = Singleton<embedding::EmbeddingManager>::Get()->GetEmbeddingState(
        embedding_name, parallel_id);

    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const std::string& state_initializer = ctx->Attr<std::string>("state_initializer");

    std::vector<EmbeddingInitializer> initializer_param;
    std::vector<int8_t> initializer_index;
    ParseInitializers(line_size, embedding_size, state_initializer,
                      ctx->Attr<std::string>("embedding_tables"), &initializer_param,
                      &initializer_index);

    const size_t param_size_bytes = initializer_param.size() * sizeof(EmbeddingInitializer);
    OF_CUDA_CHECK(hipHostMalloc(&host_initializer_param_, param_size_bytes));
    std::memcpy(host_initializer_param_, initializer_param.data(), param_size_bytes);
    OF_CUDA_CHECK(hipMalloc(&device_initializer_param_, param_size_bytes));
    OF_CUDA_CHECK(hipMemcpyAsync(device_initializer_param_, host_initializer_param_,
                                  param_size_bytes, hipMemcpyDefault,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));

    const size_t index_size_bytes = initializer_index.size() * sizeof(int8_t);
    OF_CUDA_CHECK(hipHostMalloc(&host_initializer_index_, index_size_bytes));
    std::memcpy(host_initializer_index_, initializer_index.data(), index_size_bytes);
    OF_CUDA_CHECK(hipMalloc(&device_initializer_index_, index_size_bytes));
    OF_CUDA_CHECK(hipMemcpyAsync(device_initializer_index_, host_initializer_index_,
                                  index_size_bytes, hipMemcpyDefault,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));
  }
  ~EmbeddingKernelState() override {
    CudaCurrentDeviceGuard guard(device_index_);
    OF_CUDA_CHECK(hipHostFree(host_num_keys_));
    OF_CUDA_CHECK(hipHostFree(host_initializer_param_));
    OF_CUDA_CHECK(hipFree(device_initializer_param_));
    OF_CUDA_CHECK(hipHostFree(host_initializer_index_));
    OF_CUDA_CHECK(hipFree(device_initializer_index_));
  }

  void* HostNumKeys() { return host_num_keys_; }

  embedding::KeyValueStore* KeyValueStore() { return key_value_store_; }

  embedding::EmbeddingState* EmbeddingState() { return embedding_state_; }

  const int8_t* InitializerIndex() { return device_initializer_index_; }
  const EmbeddingInitializer* Initializers() { return device_initializer_param_; }

 private:
  int device_index_;
  void* host_num_keys_;
  embedding::KeyValueStore* key_value_store_;
  embedding::EmbeddingState* embedding_state_;
  EmbeddingInitializer* host_initializer_param_;
  EmbeddingInitializer* device_initializer_param_;
  int8_t* host_initializer_index_;
  int8_t* device_initializer_index_;
};

class EmbeddingPutKernelState final : public user_op::OpKernelState {
 public:
  explicit EmbeddingPutKernelState(user_op::KernelInitContext* ctx) {
    const std::string& embedding_name = ctx->Attr<std::string>("embedding_name");
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    key_value_store_ = Singleton<embedding::EmbeddingManager>::Get()->GetKeyValueStore(
        embedding_name, parallel_id);
    uint32_t max_query_length =
        ctx->TensorDesc4ArgNameAndIndex("unique_ids", 0)->shape().elem_cnt();
    key_value_store_->ReserveQueryLength(max_query_length);
    embedding_state_ = Singleton<embedding::EmbeddingManager>::Get()->GetEmbeddingState(
        embedding_name, parallel_id);
  }
  ~EmbeddingPutKernelState() override = default;

  embedding::KeyValueStore* KeyValueStore() { return key_value_store_; }
  embedding::EmbeddingState* EmbeddingState() { return embedding_state_; }

 private:
  embedding::KeyValueStore* key_value_store_;
  embedding::EmbeddingState* embedding_state_;
};

template<typename T, typename K, typename U>
__global__ void InitValueKernel(uint64_t seed, const int32_t line_size,
                                const int32_t embedding_size,
                                const EmbeddingInitializer* initializer_param,
                                const int8_t* initializer_index, const K* unique_ids,
                                const U* table_ids, const uint32_t* num_missing_keys,
                                const uint32_t* missing_indices, T* values) {
  int64_t n = *num_missing_keys * line_size;
  CUDA_1D_KERNEL_LOOP(i, n) {
    int row = i / line_size;
    int col = i - row * line_size;
    const uint32_t index = missing_indices[row];
    const int64_t offset = index * line_size + col;
    const int32_t table_idx = table_ids[index];
    const K id = unique_ids[index];
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed, id, col, &state);
    const int32_t initializer_idx = initializer_index[table_idx * line_size + col];
    EmbeddingInitializer initializer = initializer_param[initializer_idx];
    T value;
    if (initializer.type == InitializerType::kUniform) {
      const float low = initializer.uniform_param.low;
      const float high = initializer.uniform_param.high;
      value = hiprand_uniform(&state) * (high - low) + low;
    } else if (initializer.type == InitializerType::kNormal) {
      const float mean = initializer.normal_param.mean;
      const float std = initializer.normal_param.std;
      value = hiprand_normal(&state) * std + mean;
    } else if (initializer.type == InitializerType::kConstant) {
      value = initializer.constant_param.value;
    } else if (initializer.type == InitializerType::kTruncNormal) {
      const float mean = initializer.trunc_normal_param.mean;
      const float std = initializer.trunc_normal_param.std;
      const float a = initializer.trunc_normal_param.a;
      const float b = initializer.trunc_normal_param.b;
      while (true) {
        value = hiprand_normal(&state) * std + mean;
        if (value >= a && value <= b) { break; }
        skipahead(line_size, &state);
      }
    } else {
      __trap();
    }
    values[offset] = value;
  }
}

template<typename T, typename K, typename U, typename IDX>
void LookupAndInitMissing(ep::Stream* stream, uint64_t seed, embedding::KeyValueStore* store,
                          const EmbeddingInitializer* initializer_param,
                          const int8_t* initializer_index, void* host_num_keys, uint32_t num_unique,
                          const int64_t embedding_size, const int64_t line_size,
                          const bool put_to_store, const void* unique_ids, const void* table_ids,
                          void* num_missing_ptr, void* missing_indices, void* store_values) {
  store->Get(stream, num_unique, unique_ids, store_values,
             reinterpret_cast<uint32_t*>(num_missing_ptr),
             reinterpret_cast<uint32_t*>(missing_indices));
  CHECK_GE(sizeof(IDX), sizeof(uint32_t));  // host_num_keys's buffer size is sizeof(IDX)
  OF_CUDA_CHECK(hipMemcpyAsync(host_num_keys, num_missing_ptr, sizeof(uint32_t), hipMemcpyDefault,
                                stream->As<ep::CudaStream>()->hip_stream()));
  CHECK_JUST(stream->Sync());
  uint32_t num_missing = *reinterpret_cast<uint32_t*>(host_num_keys);
  // init missing values
  if (num_missing > 0) {
    const int64_t elem_cnt = num_missing * line_size;
    const int64_t num_blocks = BlocksNum4ThreadsNum(elem_cnt);
    InitValueKernel<T, K, U>
        <<<num_blocks, kCudaThreadsNumPerBlock, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            seed, line_size, embedding_size, initializer_param, initializer_index,
            reinterpret_cast<const K*>(unique_ids), reinterpret_cast<const U*>(table_ids),
            reinterpret_cast<uint32_t*>(num_missing_ptr),
            reinterpret_cast<uint32_t*>(missing_indices), reinterpret_cast<T*>(store_values));
  }
  if (put_to_store) { store->Put(stream, num_unique, unique_ids, store_values); }
}

template<typename T, typename K, typename U, typename IDX>
void LookupAndInitMissing(ep::Stream* stream, EmbeddingKernelState<IDX>* kernel_state,
                          uint64_t seed, uint32_t num_unique, const int64_t embedding_size,
                          const int64_t line_size, const bool put_to_store, const void* unique_ids,
                          const void* table_ids, void* num_missing_ptr, void* missing_indices,
                          void* store_values) {
  embedding::KeyValueStore* store = kernel_state->KeyValueStore();
  const EmbeddingInitializer* initializer_param = kernel_state->Initializers();
  const int8_t* initializer_index = kernel_state->InitializerIndex();
  void* host_num_keys = kernel_state->HostNumKeys();
  LookupAndInitMissing<T, K, U, IDX>(stream, seed, store, initializer_param, initializer_index,
                                     host_num_keys, num_unique, embedding_size, line_size,
                                     put_to_store, unique_ids, table_ids, num_missing_ptr,
                                     missing_indices, store_values);
}

template<typename T, size_t pack_size>
struct alignas(sizeof(T) * pack_size) Pack {
  T elem[pack_size];
};

template<typename T, typename K, typename U, typename V, int pack_size>
__global__ void FusedInitSliceCast(const int32_t elem_cnt, uint64_t seed, const int32_t line_size,
                                   const int32_t embedding_size, const int32_t line_num_pack,
                                   const int32_t embedding_num_pack,
                                   const EmbeddingInitializer* initializer_param,
                                   const int8_t* initializer_index, const K* unique_ids,
                                   const U* table_ids, const uint8_t* lookup_mask,
                                   Pack<T, pack_size>* values, Pack<V, pack_size>* embeddings) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    int row = i / line_num_pack;
    int col = i - row * line_num_pack;
    Pack<T, pack_size> value_i;
    if (!lookup_mask[row]) {
      const int32_t table_idx = table_ids[row];
      const K id = unique_ids[row];
      hiprandStatePhilox4_32_10_t state;
      hiprand_init(seed, id, col, &state);
#pragma unroll
      for (int k = 0; k < pack_size; ++k) {
        const int32_t initializer_idx =
            initializer_index[table_idx * line_size + col * pack_size + k];
        EmbeddingInitializer initializer = initializer_param[initializer_idx];
        T value;
        if (initializer.type == InitializerType::kUniform) {
          const float low = initializer.uniform_param.low;
          const float high = initializer.uniform_param.high;
          value = hiprand_uniform(&state) * (high - low) + low;
        } else if (initializer.type == InitializerType::kNormal) {
          const float mean = initializer.normal_param.mean;
          const float std = initializer.normal_param.std;
          value = hiprand_normal(&state) * std + mean;
        } else if (initializer.type == InitializerType::kConstant) {
          value = initializer.constant_param.value;
        } else if (initializer.type == InitializerType::kTruncNormal) {
          const float mean = initializer.trunc_normal_param.mean;
          const float std = initializer.trunc_normal_param.std;
          const float a = initializer.trunc_normal_param.a;
          const float b = initializer.trunc_normal_param.b;
          while (true) {
            value = hiprand_normal(&state) * std + mean;
            if (value >= a && value <= b) { break; }
            skipahead(line_size, &state);
          }
        } else {
          __trap();
        }
        value_i.elem[k] = value;
      }
      values[i] = value_i;
    } else {
      value_i = values[i];
    }
    if (embeddings != nullptr && col < embedding_num_pack) {
      int64_t embedding_offset = row * embedding_num_pack + col;
      Pack<V, pack_size> embedding_i;
#pragma unroll
      for (int k = 0; k < pack_size; ++k) { embedding_i.elem[k] = static_cast<V>(value_i.elem[k]); }
      embeddings[embedding_offset] = embedding_i;
    }
  }
}

template<typename T, typename K, typename U, typename V>
void InitMissingAndSliceCast(hipStream_t hip_stream, uint32_t num_unique,
                             const int64_t embedding_size, const int64_t line_size, uint64_t seed,
                             const EmbeddingInitializer* initializer_param,
                             const int8_t* initializer_index, const void* unique_ids,
                             const void* table_ids, const uint8_t* mask, T* values_ptr,
                             V* embeddings_ptr) {
  int32_t pack_size;
  if (embedding_size % 4 == 0 && line_size % 4 == 0) {
    pack_size = 4;
  } else if (embedding_size % 2 == 0 && line_size % 2 == 0) {
    pack_size = 2;
  } else {
    pack_size = 1;
  }
  int32_t embedding_num_pack = embedding_size / pack_size;
  int32_t line_num_pack = line_size / pack_size;
  int64_t value_elem_cnt = num_unique * line_size;
  int64_t value_elem_num_pack = value_elem_cnt / pack_size;
  const int64_t num_blocks = BlocksNum4ThreadsNum(value_elem_num_pack);
  if (pack_size == 4) {
    FusedInitSliceCast<T, K, U, V, 4><<<num_blocks, kCudaThreadsNumPerBlock, 0, hip_stream>>>(
        value_elem_num_pack, seed, line_size, embedding_size, line_num_pack, embedding_num_pack,
        initializer_param, initializer_index, reinterpret_cast<const K*>(unique_ids),
        reinterpret_cast<const U*>(table_ids), mask, reinterpret_cast<Pack<T, 4>*>(values_ptr),
        reinterpret_cast<Pack<V, 4>*>(embeddings_ptr));
  } else if (pack_size == 2) {
    FusedInitSliceCast<T, K, U, V, 2><<<num_blocks, kCudaThreadsNumPerBlock, 0, hip_stream>>>(
        value_elem_num_pack, seed, line_size, embedding_size, line_num_pack, embedding_num_pack,
        initializer_param, initializer_index, reinterpret_cast<const K*>(unique_ids),
        reinterpret_cast<const U*>(table_ids), mask, reinterpret_cast<Pack<T, 2>*>(values_ptr),
        reinterpret_cast<Pack<V, 2>*>(embeddings_ptr));
  } else {
    FusedInitSliceCast<T, K, U, V, 1><<<num_blocks, kCudaThreadsNumPerBlock, 0, hip_stream>>>(
        value_elem_num_pack, seed, line_size, embedding_size, line_num_pack, embedding_num_pack,
        initializer_param, initializer_index, reinterpret_cast<const K*>(unique_ids),
        reinterpret_cast<const U*>(table_ids), mask, reinterpret_cast<Pack<T, 1>*>(values_ptr),
        reinterpret_cast<Pack<V, 1>*>(embeddings_ptr));
  }
}

template<typename T, typename K, typename U, typename IDX>
void LookupAndFusedInitMissingSliceCast(ep::Stream* stream, EmbeddingKernelState<IDX>* kernel_state,
                                        uint64_t seed, uint32_t num_unique,
                                        const int64_t embedding_size, const int64_t line_size,
                                        DataType value_dtype, DataType embedding_dtype,
                                        const void* unique_ids, const void* table_ids,
                                        uint8_t* lookup_mask_ptr, void* values_ptr,
                                        void* embeddings_ptr) {
  embedding::KeyValueStore* store = kernel_state->KeyValueStore();
  const EmbeddingInitializer* initializer_param = kernel_state->Initializers();
  const int8_t* initializer_index = kernel_state->InitializerIndex();
  hipStream_t hip_stream = stream->As<ep::CudaStream>()->hip_stream();
  store->Get(stream, num_unique, unique_ids, values_ptr, lookup_mask_ptr);
  if (embedding_dtype == value_dtype) {
    InitMissingAndSliceCast<T, K, U, T>(
        hip_stream, num_unique, embedding_size, line_size, seed, initializer_param,
        initializer_index, reinterpret_cast<const K*>(unique_ids),
        reinterpret_cast<const U*>(table_ids), lookup_mask_ptr, reinterpret_cast<T*>(values_ptr),
        reinterpret_cast<T*>(embeddings_ptr));
  } else if (embedding_dtype == DataType::kFloat16) {
    InitMissingAndSliceCast<T, K, U, half>(
        hip_stream, num_unique, embedding_size, line_size, seed, initializer_param,
        initializer_index, reinterpret_cast<const K*>(unique_ids),
        reinterpret_cast<const U*>(table_ids), lookup_mask_ptr, reinterpret_cast<T*>(values_ptr),
        reinterpret_cast<half*>(embeddings_ptr));
  } else {
    UNIMPLEMENTED() << "Unimplemented data_type " << embedding_dtype;
  }
}

template<typename T, typename U>
__global__ void Copy2D(int64_t out_elem_cnt, const int32_t in_cols, const int32_t out_cols,
                       const T* in, U* out) {
  CUDA_1D_KERNEL_LOOP(i, out_elem_cnt) {
    const int32_t row = i / out_cols;
    const int32_t col = i - row * out_cols;
    const int64_t in_offset = row * in_cols + col;
    out[i] = static_cast<U>(in[in_offset]);
  }
}

template<typename T>
void CopyValuesToEmbeddings(ep::Stream* stream, int64_t num_unique, const int32_t embedding_size,
                            const int32_t value_size, const DataType value_dtype,
                            const DataType embedding_dtype, const T* values, void* embeddings) {
  bool need_cast = (value_dtype != embedding_dtype);
  bool need_copy_nd = (embedding_size != value_size);
  CHECK(need_cast || need_copy_nd);
  if (need_cast && !need_copy_nd) {
    const int64_t cast_elem_count = num_unique * embedding_size;
    std::unique_ptr<ep::primitive::Cast> cast_primitive =
        ep::primitive::NewPrimitive<ep::primitive::CastFactory>(DeviceType::kCUDA, value_dtype,
                                                                embedding_dtype);
    cast_primitive->Launch(stream, values, embeddings, cast_elem_count);
  } else if (!need_cast && need_copy_nd) {
    const int32_t ndims = 2;
    DimVector src_pos_vec(ndims, 0);
    DimVector dst_pos_vec(ndims, 0);
    DimVector src_shape = {num_unique, value_size};
    DimVector dst_shape = {num_unique, embedding_size};
    DimVector extent_shape = {num_unique, embedding_size};
    std::unique_ptr<ep::primitive::CopyNd> copy_nd_primitive =
        ep::primitive::NewPrimitive<ep::primitive::CopyNdFactory>(DeviceType::kCUDA, ndims);
    CHECK(copy_nd_primitive);
    copy_nd_primitive->Launch(stream, value_dtype, ndims, embeddings, dst_shape.data(),
                              dst_pos_vec.data(), values, src_shape.data(), src_pos_vec.data(),
                              extent_shape.data());
  } else {
    const int64_t embedding_elem_cnt = num_unique * embedding_size;
    if (embedding_dtype == DataType::kFloat16) {
      Copy2D<T, half><<<BlocksNum4ThreadsNum(embedding_elem_cnt), kCudaThreadsNumPerBlock, 0,
                        stream->As<ep::CudaStream>()->hip_stream()>>>(
          embedding_elem_cnt, value_size, embedding_size, values,
          reinterpret_cast<half*>(embeddings));
    } else {
      UNIMPLEMENTED();
    }
  }
}

template<typename T, bool is_prefetch>
user_op::InferTmpSizeFn GenEmbeddingInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    size_t total_buffer_size = 0;
    if (embedding::UseDynamicMemoryAllocation()) { return total_buffer_size; }
    const user_op::TensorDesc& unique_ids = ctx->InputTensorDesc("unique_ids", 0);
    int64_t num_ids = unique_ids.shape().elem_cnt();
    size_t num_missing_size = GetCudaAlignedSize(sizeof(uint32_t));
    size_t missing_indices_size = GetCudaAlignedSize(num_ids * sizeof(uint32_t));
    size_t value_buffer_size;
    if (is_prefetch) {
      size_t value_byte_size = ctx->Attr<int64_t>("line_size") * sizeof(T);
      value_buffer_size = GetCudaAlignedSize(num_ids * value_byte_size);
    } else {
      value_buffer_size = 0;
    }
    total_buffer_size = num_missing_size + missing_indices_size + value_buffer_size;
    return total_buffer_size;
  };
}

class IdShuffleCopyOutKernelState final : public user_op::OpKernelState {
 public:
  explicit IdShuffleCopyOutKernelState(user_op::KernelInitContext* ctx) {
    const std::string& embedding_name = ctx->Attr<std::string>("embedding_name");
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    embedding_state_ = Singleton<embedding::EmbeddingManager>::Get()->GetEmbeddingState(
        embedding_name, parallel_id);
  }
  ~IdShuffleCopyOutKernelState() override = default;

  embedding::EmbeddingState* EmbeddingState() { return embedding_state_; }

 private:
  embedding::EmbeddingState* embedding_state_;
};

template<typename K, typename U, typename IDX>
struct IdShuffleCopyOutParam {
  uint32_t final_num_unique_ids;
  const K* cur_rank_unique_ids;
  K* out_cur_rank_unique_ids;
  const U* cur_rank_unique_table_ids;
  U* out_cur_rank_unique_table_ids;
  uint32_t cur_rank_num_ids;
  const IDX* cur_rank_inverse_indices;
  IDX* out_cur_rank_inverse_indices;
  uint32_t num_ids;
  const IDX* inverse_unique_partition_indices;
  IDX* out_inverse_unique_partition_indices;
  uint32_t num_unique_matrix_cnt;
  const IDX* num_unique_matrix;
  IDX* out_num_unique_matrix;
  const IDX* cur_rank_num_unique;
  IDX* out_cur_rank_num_unique;
};

template<typename K, typename U, typename IDX>
__global__ void CopyGpu(IdShuffleCopyOutParam<K, U, IDX> param) {
  CUDA_1D_KERNEL_LOOP_T(uint32_t, i, param.final_num_unique_ids) {
    param.out_cur_rank_unique_ids[i] = param.cur_rank_unique_ids[i];
    param.out_cur_rank_unique_table_ids[i] = param.cur_rank_unique_table_ids[i];
  }
  CUDA_1D_KERNEL_LOOP_T(uint32_t, i, param.cur_rank_num_ids) {
    param.out_cur_rank_inverse_indices[i] = param.cur_rank_inverse_indices[i];
  }
  CUDA_1D_KERNEL_LOOP_T(uint32_t, i, param.num_ids) {
    param.out_inverse_unique_partition_indices[i] = param.inverse_unique_partition_indices[i];
  }
  CUDA_1D_KERNEL_LOOP_T(uint32_t, i, param.num_unique_matrix_cnt) {
    param.out_num_unique_matrix[i] = param.num_unique_matrix[i];
  }
  if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
    *param.out_cur_rank_num_unique = *param.cur_rank_num_unique;
  }
}

}  // namespace

template<typename T, typename K, typename U, typename IDX>
class EmbeddingPrefetchKernel final : public user_op::OpKernel {
 public:
  EmbeddingPrefetchKernel() : current_iter_(0){};
  ~EmbeddingPrefetchKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingKernelState<IDX>>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingKernelState<IDX>*>(state);
    CHECK(kernel_state != nullptr);
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    std::unique_ptr<embedding::TmpBufferAllocator> allocator =
        embedding_state->NewTmpBufferAllocator(ctx);
    uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* table_ids = ctx->Tensor4ArgNameAndIndex("table_ids", 0);
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const int64_t seed = ctx->Attr<int64_t>("seed");
    void* num_missing_ptr;
    allocator->Allocate(&num_missing_ptr, sizeof(uint32_t));
    void* missing_indices_ptr;
    allocator->Allocate(&missing_indices_ptr, num_unique * sizeof(uint32_t));
    void* values_ptr;
    allocator->Allocate(&values_ptr, num_unique * line_size * sizeof(T));
    LookupAndInitMissing<T, K, U, IDX>(
        ctx->stream(), kernel_state, seed, num_unique, embedding_size, line_size, true,
        unique_ids->dptr(), table_ids->dptr(), num_missing_ptr, missing_indices_ptr, values_ptr);
    allocator->Free(num_missing_ptr);
    allocator->Free(missing_indices_ptr);
    allocator->Free(values_ptr);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define EMBEDDING_DATA_TYPE_SEQ OF_PP_MAKE_TUPLE_SEQ(float, DataType::kFloat)

#define ID_DATA_TYPE_SEQ                            \
  OF_PP_MAKE_TUPLE_SEQ(uint32_t, DataType::kUInt32) \
  OF_PP_MAKE_TUPLE_SEQ(uint64_t, DataType::kUInt64) \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32)   \
  OF_PP_MAKE_TUPLE_SEQ(int64_t, DataType::kInt64)

#define TABLE_ID_DATA_TYPE_SEQ                      \
  OF_PP_MAKE_TUPLE_SEQ(uint8_t, DataType::kUInt8)   \
  OF_PP_MAKE_TUPLE_SEQ(uint32_t, DataType::kUInt32) \
  OF_PP_MAKE_TUPLE_SEQ(uint64_t, DataType::kUInt64) \
  OF_PP_MAKE_TUPLE_SEQ(int8_t, DataType::kInt8)     \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32)   \
  OF_PP_MAKE_TUPLE_SEQ(int64_t, DataType::kInt64)

#define IDX_DATA_TYPE_SEQ                           \
  OF_PP_MAKE_TUPLE_SEQ(uint32_t, DataType::kUInt32) \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32)

#define REGISTER_CUDA_EMBEDDING_PREFETCH_KERNEL(t_dtype_pair, k_dtype_pair, table_dtype_pair,  \
                                                idx_dtype_pair)                                \
  REGISTER_USER_KERNEL("embedding_prefetch")                                                   \
      .SetCreateFn<EmbeddingPrefetchKernel<                                                    \
          OF_PP_PAIR_FIRST(t_dtype_pair), OF_PP_PAIR_FIRST(k_dtype_pair),                      \
          OF_PP_PAIR_FIRST(table_dtype_pair), OF_PP_PAIR_FIRST(idx_dtype_pair)>>()             \
      .SetIsMatchedHob(                                                                        \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                      \
          && (user_op::HobDataType("unique_ids", 0) == OF_PP_PAIR_SECOND(k_dtype_pair))        \
          && (user_op::HobDataType("table_ids", 0) == OF_PP_PAIR_SECOND(table_dtype_pair))     \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))) \
      .SetInferTmpSizeFn(GenEmbeddingInferTmpSizeFn<OF_PP_PAIR_FIRST(t_dtype_pair), true>());

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_EMBEDDING_PREFETCH_KERNEL, EMBEDDING_DATA_TYPE_SEQ,
                                 ID_DATA_TYPE_SEQ, TABLE_ID_DATA_TYPE_SEQ, IDX_DATA_TYPE_SEQ)

template<typename T, typename K, typename U, typename IDX>
class EmbeddingLookupKernel final : public user_op::OpKernel {
 public:
  EmbeddingLookupKernel() : current_iter_(0){};
  ~EmbeddingLookupKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingKernelState<IDX>>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingKernelState<IDX>*>(state);
    CHECK(kernel_state != nullptr);
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    std::unique_ptr<embedding::TmpBufferAllocator> allocator =
        embedding_state->NewTmpBufferAllocator(ctx);
    embedding_state->OnEmbeddingLookupStart(ctx, current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* table_ids = ctx->Tensor4ArgNameAndIndex("table_ids", 0);
    user_op::Tensor* unique_values = ctx->Tensor4ArgNameAndIndex("unique_values", 0);
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const bool has_output_embeddings = ctx->has_output("embeddings", 0);
    const int64_t seed = ctx->Attr<int64_t>("seed");
    uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    void* values_ptr = embedding_state->LookupUniqueValues(current_iter_);
    if (has_output_embeddings && kernel_state->KeyValueStore()->IsFusionSupported()) {
      void* embeddings_ptr = embedding_state->LookupEmbeddings(current_iter_);
      user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
      void* lookup_mask_ptr;
      allocator->Allocate(&lookup_mask_ptr, num_unique * sizeof(uint8_t));
      LookupAndFusedInitMissingSliceCast<T, K, U, IDX>(
          ctx->stream(), kernel_state, seed, num_unique, embedding_size, line_size,
          unique_values->data_type(), embeddings->data_type(), unique_ids->dptr(),
          table_ids->dptr(), reinterpret_cast<uint8_t*>(lookup_mask_ptr), values_ptr,
          embeddings_ptr);
      allocator->Free(lookup_mask_ptr);
    } else {
      void* num_missing_ptr;
      allocator->Allocate(&num_missing_ptr, sizeof(uint32_t));
      void* missing_indices_ptr;
      allocator->Allocate(&missing_indices_ptr, num_unique * sizeof(uint32_t));
      LookupAndInitMissing<T, K, U, IDX>(
          ctx->stream(), kernel_state, seed, num_unique, embedding_size, line_size, false,
          unique_ids->dptr(), table_ids->dptr(), num_missing_ptr, missing_indices_ptr, values_ptr);
      allocator->Free(num_missing_ptr);
      allocator->Free(missing_indices_ptr);
      if (has_output_embeddings) {
        void* embeddings_ptr = embedding_state->LookupEmbeddings(current_iter_);
        user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
        CopyValuesToEmbeddings<T>(ctx->stream(), num_unique, embedding_size, line_size,
                                  unique_values->data_type(), embeddings->data_type(),
                                  reinterpret_cast<T*>(values_ptr), embeddings_ptr);
      }
    }
    embedding_state->OnEmbeddingLookupEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL(t_dtype_pair, k_dtype_pair, table_dtype_pair,    \
                                              idx_dtype_pair)                                  \
  REGISTER_USER_KERNEL("embedding_lookup")                                                     \
      .SetCreateFn<EmbeddingLookupKernel<                                                      \
          OF_PP_PAIR_FIRST(t_dtype_pair), OF_PP_PAIR_FIRST(k_dtype_pair),                      \
          OF_PP_PAIR_FIRST(table_dtype_pair), OF_PP_PAIR_FIRST(idx_dtype_pair)>>()             \
      .SetIsMatchedHob(                                                                        \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                      \
          && (user_op::HobDataType("unique_values", 0) == OF_PP_PAIR_SECOND(t_dtype_pair))     \
          && (user_op::HobDataType("unique_ids", 0) == OF_PP_PAIR_SECOND(k_dtype_pair))        \
          && (user_op::HobDataType("table_ids", 0) == OF_PP_PAIR_SECOND(table_dtype_pair))     \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))) \
      .SetInferTmpSizeFn(GenEmbeddingInferTmpSizeFn<OF_PP_PAIR_FIRST(t_dtype_pair), false>());

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL, EMBEDDING_DATA_TYPE_SEQ,
                                 ID_DATA_TYPE_SEQ, TABLE_ID_DATA_TYPE_SEQ, IDX_DATA_TYPE_SEQ)

template<typename IDX>
class EmbeddingPutKernel final : public user_op::OpKernel {
 public:
  EmbeddingPutKernel() : current_iter_(0){};
  ~EmbeddingPutKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingPutKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingPutKernelState*>(state);
    CHECK(kernel_state != nullptr);
    embedding::KeyValueStore* store = kernel_state->KeyValueStore();
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingPutStart(ctx, current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* unique_embeddings = ctx->Tensor4ArgNameAndIndex("unique_embeddings", 0);
    uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    store->Put(ctx->stream(), num_unique, unique_ids->dptr(),
               embedding_state->EmbeddingPutUniqueEmbeddings(current_iter_));
    embedding_state->OnEmbeddingPutEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define REGISTER_CUDA_EMBEDDING_PUT_KERNEL(dtype, typeproto)           \
  REGISTER_USER_KERNEL("embedding_put")                                \
      .SetCreateFn<EmbeddingPutKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("num_unique_ids", 0) == typeproto));

OF_PP_FOR_EACH_TUPLE(REGISTER_CUDA_EMBEDDING_PUT_KERNEL, IDX_DATA_TYPE_SEQ)

template<typename IDX>
class OneEmbeddingFusedSgdUpdatePutKernel final : public user_op::OpKernel {
 public:
  OneEmbeddingFusedSgdUpdatePutKernel() : current_iter_(0){};
  ~OneEmbeddingFusedSgdUpdatePutKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingPutKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingPutKernelState*>(state);
    CHECK(kernel_state != nullptr);
    embedding::KeyValueStore* store = kernel_state->KeyValueStore();
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingFusedUpdatePutStart(ctx, current_iter_);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
    const float* learning_rate_ptr = learning_rate->dptr<float>();
    const auto scale = ctx->Attr<double>("scale");
    uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    store->FusedHalfUpdatePut(
        ctx->stream(), num_unique, unique_ids->dptr(),
        embedding_state->EmbeddingFusedUpdatePutUniqueEmbeddings(current_iter_),
        embedding_grad->dptr(), learning_rate_ptr, scale);
    embedding_state->OnEmbeddingFusedUpdatePutEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define REGISTER_CUDA_ONE_EMBEDDING_FUSED_SGD_UPDATE_PUT_KERNEL(dtype, typeproto)            \
  REGISTER_USER_KERNEL("one_embedding_fused_sgd_update_put")                                 \
      .SetCreateFn<OneEmbeddingFusedSgdUpdatePutKernel<dtype>>()                             \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                       \
                       && (user_op::HobDataType("num_unique_ids", 0) == typeproto)           \
                       && (user_op::HobDataType("unique_embeddings", 0) == DataType::kFloat) \
                       && (user_op::HobDataType("embedding_grad", 0) == DataType::kFloat16));

OF_PP_FOR_EACH_TUPLE(REGISTER_CUDA_ONE_EMBEDDING_FUSED_SGD_UPDATE_PUT_KERNEL, IDX_DATA_TYPE_SEQ)

template<typename K, typename U, typename IDX>
class IdShuffleCopyOutKernel final : public user_op::OpKernel {
 public:
  IdShuffleCopyOutKernel() : current_iter_(0){};
  ~IdShuffleCopyOutKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<IdShuffleCopyOutKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<IdShuffleCopyOutKernelState*>(state);
    CHECK(kernel_state != nullptr);
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    const uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    const std::vector<uint32_t>& num_unique_matrix_vec =
        embedding_state->GetIdNumUniqueMatrix(current_iter_);
    uint32_t cur_rank_num_ids = 0;
    for (int64_t i = 0; i < parallel_num; ++i) {
      cur_rank_num_ids += num_unique_matrix_vec.at(i * parallel_num + parallel_id);
    }
    IdShuffleCopyOutParam<K, U, IDX> param;
    param.final_num_unique_ids = num_unique;
    param.cur_rank_unique_ids =
        reinterpret_cast<const K*>(ctx->Tensor4ArgNameAndIndex("cur_rank_unique_ids", 0)->dptr());
    param.out_cur_rank_unique_ids =
        reinterpret_cast<K*>(ctx->Tensor4ArgNameAndIndex("out_cur_rank_unique_ids", 0)->mut_dptr());
    param.cur_rank_unique_table_ids = reinterpret_cast<const U*>(
        ctx->Tensor4ArgNameAndIndex("cur_rank_unique_table_ids", 0)->dptr());
    param.out_cur_rank_unique_table_ids = reinterpret_cast<U*>(
        ctx->Tensor4ArgNameAndIndex("out_cur_rank_unique_table_ids", 0)->mut_dptr());
    param.cur_rank_num_ids = cur_rank_num_ids;
    param.cur_rank_inverse_indices = reinterpret_cast<const IDX*>(
        ctx->Tensor4ArgNameAndIndex("cur_rank_inverse_indices", 0)->dptr());
    param.out_cur_rank_inverse_indices = reinterpret_cast<IDX*>(
        ctx->Tensor4ArgNameAndIndex("out_cur_rank_inverse_indices", 0)->mut_dptr());
    param.num_ids =
        ctx->Tensor4ArgNameAndIndex("inverse_unique_partition_indices", 0)->shape_view().elem_cnt();
    param.inverse_unique_partition_indices = reinterpret_cast<const IDX*>(
        ctx->Tensor4ArgNameAndIndex("inverse_unique_partition_indices", 0)->dptr());
    param.out_inverse_unique_partition_indices = reinterpret_cast<IDX*>(
        ctx->Tensor4ArgNameAndIndex("out_inverse_unique_partition_indices", 0)->mut_dptr());
    param.num_unique_matrix_cnt = parallel_num * parallel_num;
    param.num_unique_matrix =
        reinterpret_cast<const IDX*>(ctx->Tensor4ArgNameAndIndex("num_unique_matrix", 0)->dptr());
    param.out_num_unique_matrix =
        reinterpret_cast<IDX*>(ctx->Tensor4ArgNameAndIndex("out_num_unique_matrix", 0)->mut_dptr());
    param.cur_rank_num_unique =
        reinterpret_cast<const IDX*>(ctx->Tensor4ArgNameAndIndex("cur_rank_num_unique", 0)->dptr());
    param.out_cur_rank_num_unique = reinterpret_cast<IDX*>(
        ctx->Tensor4ArgNameAndIndex("out_cur_rank_num_unique", 0)->mut_dptr());

    CopyGpu<K, U, IDX><<<BlocksNum4ThreadsNum(param.num_ids), kCudaThreadsNumPerBlock, 0,
                         ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(param);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define REGISTER_CUDA_ID_SHUFFLE_COPY_OUT_KERNEL(k_dtype_pair, table_id_dtype_pair,              \
                                                 idx_dtype_pair)                                 \
  REGISTER_USER_KERNEL("id_shuffle_copy_out")                                                    \
      .SetCreateFn<IdShuffleCopyOutKernel<OF_PP_PAIR_FIRST(k_dtype_pair),                        \
                                          OF_PP_PAIR_FIRST(table_id_dtype_pair),                 \
                                          OF_PP_PAIR_FIRST(idx_dtype_pair)>>()                   \
      .SetIsMatchedHob(                                                                          \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                        \
          && (user_op::HobDataType("cur_rank_unique_ids", 0) == OF_PP_PAIR_SECOND(k_dtype_pair)) \
          && (user_op::HobDataType("cur_rank_unique_table_ids", 0)                               \
              == OF_PP_PAIR_SECOND(table_id_dtype_pair))                                         \
          && (user_op::HobDataType("num_unique_matrix", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair)));

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_ID_SHUFFLE_COPY_OUT_KERNEL, ID_DATA_TYPE_SEQ,
                                 TABLE_ID_DATA_TYPE_SEQ, IDX_DATA_TYPE_SEQ)

enum class FusedEmbeddingBufferType {
  // id shuffle
  kNumPartitionedUnique = 0,
  kPartitionedUniqueIds,
  kReceivedIds,
  kTableIds,
  kPartitionedUniqueTableIds,
  kReceivedTableIds,
  kWorkspace,
  kNumUniqueMatrix,
  kInverseUniquePartitionIndices,
  kCurRankNumUnique,
  kCurRankUniqueIds,
  kCurRankUniqueTableIds,
  kCurRankInverseIndices,
  // embedding lookup
  kNumMissing,
  kMissingIndices,
  kCurRankUniqueValues,
  kCurRankUniqueEmbeddings,
  // embedding shuffle
  kReverseUniqueCurRankEmbeddings,
  kReceivedEmbeddings,
  kMaxType
};

template<typename K, typename U, typename IDX>
class FusedEmbeddingTmpBufferManager final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(FusedEmbeddingTmpBufferManager);
  FusedEmbeddingTmpBufferManager(void* ptr, const int64_t num_ids, const int64_t parallel_num,
                                 bool need_process_table_ids, int64_t line_size,
                                 int64_t embedding_size, bool need_unique_values,
                                 bool need_embeddings, DataType value_dtype,
                                 DataType embedding_dtype)
      : offset_(0),
        offsets_(static_cast<size_t>(FusedEmbeddingBufferType::kMaxType), -1),
        sizes_(static_cast<size_t>(FusedEmbeddingBufferType::kMaxType)),
        ptr_(ptr) {
    // id shuffle
    const int64_t num_table_ids = need_process_table_ids ? num_ids : 0;
    const size_t table_ids_bytes = need_process_table_ids ? num_ids * sizeof(U) : 0;
    AllocBuffer(FusedEmbeddingBufferType::kNumPartitionedUnique, parallel_num * sizeof(IDX));
    size_t partitioned_ids_bytes = parallel_num * num_ids * sizeof(K);
    AllocBuffer(FusedEmbeddingBufferType::kPartitionedUniqueIds, partitioned_ids_bytes);
    AllocBuffer(FusedEmbeddingBufferType::kReceivedIds, partitioned_ids_bytes);
    AllocBuffer(FusedEmbeddingBufferType::kTableIds, table_ids_bytes);
    size_t partitioned_table_ids_bytes = parallel_num * num_table_ids * sizeof(U);
    AllocBuffer(FusedEmbeddingBufferType::kPartitionedUniqueTableIds, partitioned_table_ids_bytes);
    AllocBuffer(FusedEmbeddingBufferType::kReceivedTableIds, partitioned_table_ids_bytes);
    const size_t hash_table_capacity = parallel_num * num_ids;
    AllocBuffer(FusedEmbeddingBufferType::kWorkspace,
                hash_table_capacity * sizeof(data_shuffle::TableEntry<K>));
    size_t num_unique_matrix_bytes = parallel_num * parallel_num * sizeof(IDX);
    AllocBuffer(FusedEmbeddingBufferType::kNumUniqueMatrix, num_unique_matrix_bytes);
    size_t inverse_unique_partition_indices_bytes = num_ids * sizeof(IDX);
    AllocBuffer(FusedEmbeddingBufferType::kInverseUniquePartitionIndices,
                inverse_unique_partition_indices_bytes);
    size_t cur_rank_num_ids = parallel_num * num_ids;
    size_t cur_rank_num_table_ids = cur_rank_num_ids;
    size_t cur_rank_num_unique_bytes = sizeof(uint32_t);
    AllocBuffer(FusedEmbeddingBufferType::kCurRankNumUnique, cur_rank_num_unique_bytes);
    size_t cur_rank_unique_ids_bytes = cur_rank_num_ids * sizeof(K);
    AllocBuffer(FusedEmbeddingBufferType::kCurRankUniqueIds, cur_rank_unique_ids_bytes);
    size_t cur_rank_unique_table_ids_bytes = cur_rank_num_table_ids * sizeof(U);
    AllocBuffer(FusedEmbeddingBufferType::kCurRankUniqueTableIds, cur_rank_unique_table_ids_bytes);
    size_t cur_rank_inverse_indices_bytes = cur_rank_num_ids * sizeof(IDX);
    AllocBuffer(FusedEmbeddingBufferType::kCurRankInverseIndices, cur_rank_inverse_indices_bytes);
    // embedding lookup
    size_t num_missing_bytes = sizeof(uint32_t);
    AllocBuffer(FusedEmbeddingBufferType::kNumMissing, num_missing_bytes);
    size_t missing_indices_bytes = cur_rank_num_ids * sizeof(uint32_t);
    AllocBuffer(FusedEmbeddingBufferType::kMissingIndices, missing_indices_bytes);
    if (need_unique_values) {
      size_t cur_rank_unique_values_bytes =
          cur_rank_num_ids * line_size * GetSizeOfDataType(value_dtype);
      AllocBuffer(FusedEmbeddingBufferType::kCurRankUniqueValues, cur_rank_unique_values_bytes);
    }
    if (need_embeddings) {
      size_t cur_rank_unique_embeddings_bytes =
          cur_rank_num_ids * embedding_size * GetSizeOfDataType(embedding_dtype);
      AllocBuffer(FusedEmbeddingBufferType::kCurRankUniqueEmbeddings,
                  cur_rank_unique_embeddings_bytes);
    }
    // embedding shuffle
    size_t reverse_unique_cur_rank_embeddings_bytes =
        cur_rank_num_ids * embedding_size * GetSizeOfDataType(embedding_dtype);
    AllocBuffer(FusedEmbeddingBufferType::kReverseUniqueCurRankEmbeddings,
                reverse_unique_cur_rank_embeddings_bytes);
    size_t received_embeddings_bytes =
        cur_rank_num_ids * embedding_size * GetSizeOfDataType(embedding_dtype);
    AllocBuffer(FusedEmbeddingBufferType::kReceivedEmbeddings, received_embeddings_bytes);
  }

  template<typename T = void>
  T* Ptr(FusedEmbeddingBufferType type) const {
    CHECK(ptr_ != nullptr);
    int64_t offset = offsets_.at(static_cast<size_t>(type));
    CHECK_NE(offset, -1);
    return reinterpret_cast<T*>(reinterpret_cast<char*>(ptr_) + offset);
  }

  int64_t Size(FusedEmbeddingBufferType type) const { return sizes_.at(static_cast<size_t>(type)); }

  size_t TotalBufferSize() const { return offset_; }

 private:
  void AllocBuffer(FusedEmbeddingBufferType type, size_t size) {
    const size_t type_id = static_cast<size_t>(type);
    CHECK_EQ(offsets_.at(type_id), -1);
    offsets_.at(type_id) = offset_;
    sizes_.at(type_id) = size;
    offset_ += GetCudaAlignedSize(size);
  }
  size_t offset_;
  std::vector<int64_t> offsets_;
  std::vector<int64_t> sizes_;
  void* ptr_;
};

void MakeConstantInitializerAttr(const int64_t embedding_size, const int64_t line_size,
                                 const std::vector<float>& values, std::string* initializer_attr) {
  if (embedding_size == line_size) { return; }
  const int32_t num_states = line_size / embedding_size - 1;
  CHECK_GT(num_states, 0) << "num_states " << num_states;
  CHECK(values.size() == 0 || num_states == values.size())
      << "must set " << num_states << " optimizer states init value, but get " << values.size();
  nlohmann::json initializers;
  for (int32_t i = 0; i < num_states; ++i) {
    nlohmann::json initializer;
    initializer["type"] = "constant";
    const float initial_value = values.size() > 0 ? values.at(i) : 0.0;
    initializer["value"] = initial_value;
    initializers.push_back(initializer);
  }
  *initializer_attr = initializers.dump();
}

template<typename IDX>
class OneEmbeddingFusedLookupKernelState final : public user_op::OpKernelState {
 public:
  explicit OneEmbeddingFusedLookupKernelState(user_op::KernelInitContext* ctx)
      : device_index_(-1),
        stream_name_(EagerNcclCommMgr::kDefaultStreamName),
        parallel_desc_(ctx->parallel_desc()) {
    OF_CUDA_CHECK(hipGetDevice(&device_index_));
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    OF_CUDA_CHECK(hipHostMalloc(&host_num_keys_, sizeof(IDX)));
    OF_CUDA_CHECK(
        hipHostMalloc(&host_num_unique_matrix_, parallel_num * parallel_num * sizeof(IDX)));
    const std::string& embedding_name = ctx->Attr<std::string>("embedding_name");
    key_value_store_ = Singleton<embedding::EmbeddingManager>::Get()->GetKeyValueStore(
        embedding_name, parallel_id);
    uint32_t max_query_length =
        ctx->TensorDesc4ArgNameAndIndex("ids", 0)->shape().elem_cnt() * parallel_num;
    key_value_store_->ReserveQueryLength(max_query_length);

    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    // Note(guoran): This op have no optimizer info, so set embedding states initializer constant
    // 0, which may make error in optimizer with initial_accumulator_value like adagrad and ftrl.
    std::string state_initializer;
    MakeConstantInitializerAttr(embedding_size, line_size, {}, &state_initializer);

    std::vector<EmbeddingInitializer> initializer_param;
    std::vector<int8_t> initializer_index;
    ParseInitializers(line_size, embedding_size, state_initializer,
                      ctx->Attr<std::string>("embedding_tables"), &initializer_param,
                      &initializer_index);

    const size_t param_size_bytes = initializer_param.size() * sizeof(EmbeddingInitializer);
    OF_CUDA_CHECK(hipHostMalloc(&host_initializer_param_, param_size_bytes));
    std::memcpy(host_initializer_param_, initializer_param.data(), param_size_bytes);
    OF_CUDA_CHECK(hipMalloc(&device_initializer_param_, param_size_bytes));
    OF_CUDA_CHECK(hipMemcpyAsync(device_initializer_param_, host_initializer_param_,
                                  param_size_bytes, hipMemcpyDefault,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));

    const size_t index_size_bytes = initializer_index.size() * sizeof(int8_t);
    OF_CUDA_CHECK(hipHostMalloc(&host_initializer_index_, index_size_bytes));
    std::memcpy(host_initializer_index_, initializer_index.data(), index_size_bytes);
    OF_CUDA_CHECK(hipMalloc(&device_initializer_index_, index_size_bytes));
    OF_CUDA_CHECK(hipMemcpyAsync(device_initializer_index_, host_initializer_index_,
                                  index_size_bytes, hipMemcpyDefault,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));
  }
  ~OneEmbeddingFusedLookupKernelState() override {
    CudaCurrentDeviceGuard guard(device_index_);
    OF_CUDA_CHECK(hipHostFree(host_num_keys_));
    OF_CUDA_CHECK(hipHostFree(host_num_unique_matrix_));
    OF_CUDA_CHECK(hipHostFree(host_initializer_param_));
    OF_CUDA_CHECK(hipFree(device_initializer_param_));
    OF_CUDA_CHECK(hipHostFree(host_initializer_index_));
    OF_CUDA_CHECK(hipFree(device_initializer_index_));
  }

  ncclComm_t comm() { return GetOrCreate().comm; }

  IDX* HostNumUniqueMatrix() { return host_num_unique_matrix_; }

  IDX* HostNumKeys() { return host_num_keys_; }

  embedding::KeyValueStore* KeyValueStore() { return key_value_store_; }

  const int8_t* InitializerIndex() { return device_initializer_index_; }
  const EmbeddingInitializer* Initializers() { return device_initializer_param_; }

 private:
  struct Comm {
    Comm(ncclComm_t comm) : comm(comm) {}
    ncclComm_t comm;
  };

  const Comm& GetOrCreate() {
    if (!comm_) { Init(); }
    return *comm_;
  }

  void Init() {
    std::set<std::pair<int64_t, int64_t>> device_set;
    for (int64_t parallel_id = 0; parallel_id < parallel_desc_.parallel_num(); ++parallel_id) {
      int64_t machine_id = CHECK_JUST(parallel_desc_.MachineId4ParallelId(parallel_id));
      int64_t device_id = CHECK_JUST(parallel_desc_.DeviceId4ParallelId(parallel_id));
      device_set.emplace(std::make_pair(machine_id, device_id));
    }
    EagerNcclCommMgr* comm_mgr = CHECK_NOTNULL(Singleton<EagerNcclCommMgr>::Get());
    ncclComm_t comm;
    comm = comm_mgr->GetCommForDeviceAndStreamName(device_set, stream_name_);
    comm_.reset(new Comm(comm));
  }

  int device_index_;
  std::string stream_name_;
  ParallelDesc parallel_desc_;
  std::unique_ptr<Comm> comm_;
  IDX* host_num_keys_;
  IDX* host_num_unique_matrix_;
  embedding::KeyValueStore* key_value_store_;

  EmbeddingInitializer* host_initializer_param_;
  EmbeddingInitializer* device_initializer_param_;
  int8_t* host_initializer_index_;
  int8_t* device_initializer_index_;
};

template<typename T, typename K, typename U, typename IDX>
void LookupAndInitMissing(ep::Stream* stream, OneEmbeddingFusedLookupKernelState<IDX>* kernel_state,
                          uint64_t seed, uint32_t num_unique, const int64_t embedding_size,
                          const int64_t line_size, const bool put_to_store, const void* unique_ids,
                          const void* table_ids, void* num_missing_ptr, void* missing_indices,
                          void* store_values) {
  embedding::KeyValueStore* store = kernel_state->KeyValueStore();
  const EmbeddingInitializer* initializer_param = kernel_state->Initializers();
  const int8_t* initializer_index = kernel_state->InitializerIndex();
  void* host_num_keys = kernel_state->HostNumKeys();
  LookupAndInitMissing<T, K, U, IDX>(stream, seed, store, initializer_param, initializer_index,
                                     host_num_keys, num_unique, embedding_size, line_size,
                                     put_to_store, unique_ids, table_ids, num_missing_ptr,
                                     missing_indices, store_values);
}

template<typename K, typename U, typename IDX>
void SetIdShuffleDataPtrsParam(const void* ids_ptr,
                               const FusedEmbeddingTmpBufferManager<K, U, IDX>& buffer_manager,
                               data_shuffle::IdShuffleDataPtrs<K, U, IDX>* data_ptrs) {
  data_ptrs->ids_ptr = reinterpret_cast<const K*>(ids_ptr);
  data_ptrs->table_ids_ptr = buffer_manager.template Ptr<U>(FusedEmbeddingBufferType::kTableIds);
  data_ptrs->num_partitioned_unique =
      buffer_manager.template Ptr<IDX>(FusedEmbeddingBufferType::kNumPartitionedUnique);
  data_ptrs->partitioned_unique_ids =
      buffer_manager.template Ptr<K>(FusedEmbeddingBufferType::kPartitionedUniqueIds);
  data_ptrs->partitioned_unique_table_ids =
      buffer_manager.template Ptr<U>(FusedEmbeddingBufferType::kPartitionedUniqueTableIds);
  data_ptrs->workspace_ptr = buffer_manager.Ptr(FusedEmbeddingBufferType::kWorkspace);
  data_ptrs->workspace_size = buffer_manager.Size(FusedEmbeddingBufferType::kWorkspace);
  data_ptrs->received_ids = buffer_manager.template Ptr<K>(FusedEmbeddingBufferType::kReceivedIds);
  data_ptrs->received_table_ids =
      buffer_manager.template Ptr<U>(FusedEmbeddingBufferType::kReceivedTableIds);
  data_ptrs->inverse_unique_partition_indices_ptr =
      buffer_manager.template Ptr<IDX>(FusedEmbeddingBufferType::kInverseUniquePartitionIndices);
  data_ptrs->num_unique_matrix_ptr =
      buffer_manager.template Ptr<IDX>(FusedEmbeddingBufferType::kNumUniqueMatrix);
  data_ptrs->cur_rank_num_unique_ptr =
      buffer_manager.template Ptr<IDX>(FusedEmbeddingBufferType::kCurRankNumUnique);
  data_ptrs->cur_rank_unique_ids_ptr =
      buffer_manager.template Ptr<K>(FusedEmbeddingBufferType::kCurRankUniqueIds);
  data_ptrs->cur_rank_unique_table_ids_ptr =
      buffer_manager.template Ptr<U>(FusedEmbeddingBufferType::kCurRankUniqueTableIds);
  data_ptrs->cur_rank_inverse_indices_ptr =
      buffer_manager.template Ptr<IDX>(FusedEmbeddingBufferType::kCurRankInverseIndices);
}

template<typename K, typename T, typename V, typename U, typename IDX>
class OneEmbeddingFusedLookupKernel final : public user_op::OpKernel {
 public:
  OneEmbeddingFusedLookupKernel() = default;
  ~OneEmbeddingFusedLookupKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<OneEmbeddingFusedLookupKernelState<IDX>>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    // IDX type is uint32_t, table_ids type is uint8_t.
    DataType num_unique_matrix_dtype = DataType::kUInt32;
    DataType table_ids_dtype = DataType::kUInt8;
    CHECK_EQ(sizeof(IDX), GetSizeOfDataType(num_unique_matrix_dtype));
    CHECK_EQ(sizeof(U), GetSizeOfDataType(table_ids_dtype));
    auto* kernel_state = dynamic_cast<OneEmbeddingFusedLookupKernelState<IDX>*>(state);
    CHECK(kernel_state != nullptr);
    const user_op::Tensor* ids = ctx->Tensor4ArgNameAndIndex("ids", 0);
    user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
    const int32_t num_tables = ctx->Attr<int32_t>("num_tables");
    // default uint8_t as table_ids type, so num_tables can not greater than 256.
    CHECK_LE(num_tables, 256) << num_tables;
    const bool has_table_ids = ctx->has_input("table_ids", 0);
    const bool need_process_table_ids = (has_table_ids || num_tables > 1);
    const int64_t num_ids = ids->shape_view().elem_cnt();
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    hipStream_t hip_stream = ctx->stream()->As<ep::CudaStream>()->hip_stream();
    DataType value_dtype = ctx->Attr<DataType>("dtype");
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const int64_t padding_idx = ctx->Attr<int64_t>("padding_idx");
    const bool has_padding_idx = ctx->Attr<bool>("has_padding_idx");
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    bool need_unique_values = true;
    bool need_embeddings =
        (line_size != embedding_size) || (value_dtype != embeddings->data_type());
    FusedEmbeddingTmpBufferManager<K, U, IDX> buffer_manager(
        tmp_buffer->mut_dptr(), num_ids, parallel_num, need_process_table_ids, line_size,
        embedding_size, need_unique_values, need_embeddings, value_dtype, embeddings->data_type());
    CHECK_GE(tmp_buffer->shape_view().elem_cnt(), buffer_manager.TotalBufferSize());
    ncclComm_t comm = kernel_state->comm();
    IDX* host_num_unique_matrix = kernel_state->HostNumUniqueMatrix();
    IDX* host_num_keys = kernel_state->HostNumKeys();
    data_shuffle::IdShuffleDataPtrs<K, U, IDX> data_ptrs;
    SetIdShuffleDataPtrsParam(ids->dptr(), buffer_manager, &data_ptrs);
    // overwrite data_ptrs.table_ids_ptr
    if (need_process_table_ids) {
      U* tmp_table_ids_ptr = buffer_manager.template Ptr<U>(FusedEmbeddingBufferType::kTableIds);
      data_ptrs.table_ids_ptr = tmp_table_ids_ptr;
      if (has_table_ids) {
        // use table_id default data_type uint8, if has input table_ids with different data_type,
        // cast it to uint8.
        const user_op::Tensor* table_ids = ctx->Tensor4ArgNameAndIndex("table_ids", 0);
        if (table_ids->data_type() != table_ids_dtype) {
          std::unique_ptr<ep::primitive::Cast> cast_primitive =
              ep::primitive::NewPrimitive<ep::primitive::CastFactory>(
                  DeviceType::kCUDA, table_ids->data_type(), table_ids_dtype);
          cast_primitive->Launch(ctx->stream(), table_ids->dptr(), tmp_table_ids_ptr,
                                 table_ids->shape_view().elem_cnt());
        } else {
          data_ptrs.table_ids_ptr = reinterpret_cast<const U*>(table_ids->dptr());
        }
      } else {
        const int32_t num_tables = ctx->Attr<int32_t>("num_tables");
        data_shuffle::GenerateTableIds<<<BlocksNum4ThreadsNum(num_ids), kCudaThreadsNumPerBlock, 0,
                                         hip_stream>>>(num_ids, num_tables, tmp_table_ids_ptr);
      }
    } else {
      data_ptrs.table_ids_ptr = nullptr;
    }

    data_shuffle::IdShuffle(ctx->stream(), comm, data_ptrs, num_ids, parallel_id, parallel_num,
                            num_unique_matrix_dtype, ids->data_type(), table_ids_dtype,
                            need_process_table_ids, has_padding_idx, padding_idx,
                            host_num_unique_matrix, host_num_keys);
    uint32_t num_unique = *host_num_keys;

    // lookup and put, if is_full_cache, not put to store.
    uint32_t* num_missing_ptr =
        buffer_manager.template Ptr<uint32_t>(FusedEmbeddingBufferType::kNumMissing);
    uint32_t* missing_indices_ptr =
        buffer_manager.template Ptr<uint32_t>(FusedEmbeddingBufferType::kMissingIndices);
    void* values_ptr =
        buffer_manager.template Ptr<V>(FusedEmbeddingBufferType::kCurRankUniqueValues);
    T* cur_rank_embeddings_ptr =
        need_embeddings
            ? buffer_manager.template Ptr<T>(FusedEmbeddingBufferType::kCurRankUniqueEmbeddings)
            : reinterpret_cast<T*>(values_ptr);
    const bool is_full_cache = ctx->Attr<bool>("is_full_cache");
    const bool put_to_store = (!is_full_cache);
    const int64_t seed = ctx->Attr<int64_t>("seed");
    LookupAndInitMissing<V, K, U, IDX>(
        ctx->stream(), kernel_state, seed, num_unique, embedding_size, line_size, put_to_store,
        data_ptrs.cur_rank_unique_ids_ptr, data_ptrs.cur_rank_unique_table_ids_ptr, num_missing_ptr,
        missing_indices_ptr, values_ptr);
    if (need_embeddings) {
      CopyValuesToEmbeddings<V>(ctx->stream(), num_unique, embedding_size, line_size, value_dtype,
                                embeddings->data_type(), reinterpret_cast<V*>(values_ptr),
                                cur_rank_embeddings_ptr);
    }

    // embedding shuffle
    int64_t cur_rank_num_ids = 0;
    for (int64_t i = 0; i < parallel_num; ++i) {
      cur_rank_num_ids += host_num_unique_matrix[i * parallel_num + parallel_id];
    }
    int64_t unique_partitioned_num_ids = 0;
    for (int64_t i = 0; i < parallel_num; ++i) {
      unique_partitioned_num_ids += host_num_unique_matrix[parallel_id * parallel_num + i];
    }
    T* reverse_unique_cur_rank_embeddings_ptr =
        buffer_manager.template Ptr<T>(FusedEmbeddingBufferType::kReverseUniqueCurRankEmbeddings);
    T* received_embeddings_ptr =
        buffer_manager.template Ptr<T>(FusedEmbeddingBufferType::kReceivedEmbeddings);
    GatherKernelUtilImpl<DeviceType::kCUDA, T, IDX>::Forward(
        ctx->stream(), data_ptrs.cur_rank_inverse_indices_ptr, cur_rank_num_ids,
        cur_rank_embeddings_ptr, Shape({1, num_unique, embedding_size}),
        reverse_unique_cur_rank_embeddings_ptr, 0);

    data_shuffle::ShuffleEmbeddings(hip_stream, comm, parallel_id, parallel_num, num_ids,
                                    embedding_size, embeddings->data_type(), host_num_unique_matrix,
                                    reverse_unique_cur_rank_embeddings_ptr,
                                    received_embeddings_ptr);
    GatherKernelUtilImpl<DeviceType::kCUDA, T, IDX>::Forward(
        ctx->stream(), data_ptrs.inverse_unique_partition_indices_ptr, num_ids,
        received_embeddings_ptr, Shape({1, unique_partitioned_num_ids, embedding_size}),
        embeddings->mut_dptr<T>(), 0);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

auto SingleDeviceKernel() {
  return hob::make_custom("SingleDeviceKernel", [](const user_op::KernelRegContext& ctx) {
    return (ctx.parallel_ctx().parallel_num() == 1);
  });
}

// Note(guoran): Default use U type as uint8_t, IDX as uint32_t. Because table_ids is optional, so
// can not use it in hob, if has table_ids input and dtype is not uint8_t cast to uint8_t in kernel.
#define REGISTER_CUDA_ONE_EMBEDDING_FUSED_LOOKUP_KERNEL(k_dtype_pair, t_dtype_pair, v_dtype_pair) \
  REGISTER_USER_KERNEL("one_embedding_fused_lookup")                                              \
      .SetCreateFn<OneEmbeddingFusedLookupKernel<                                                 \
          OF_PP_PAIR_FIRST(k_dtype_pair), OF_PP_PAIR_FIRST(t_dtype_pair),                         \
          OF_PP_PAIR_FIRST(v_dtype_pair), uint8_t, uint32_t>>()                                   \
      .SetIsMatchedHob(                                                                           \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                         \
          && (user_op::HobDataType("ids", 0) == OF_PP_PAIR_SECOND(k_dtype_pair))                  \
          && (user_op::HobDataType("embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair))           \
          && (user_op::HobAttr<DataType>("dtype") == OF_PP_PAIR_SECOND(v_dtype_pair))             \
          && !SingleDeviceKernel())                                                               \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                         \
        const user_op::TensorDesc& ids = ctx->InputTensorDesc("ids", 0);                          \
        const user_op::TensorDesc& embeddings = ctx->OutputTensorDesc("embeddings", 0);           \
        const bool has_table_ids = ctx->has_input("table_ids", 0);                                \
        const int32_t num_tables = ctx->Attr<int32_t>("num_tables");                              \
        const bool need_process_table_ids = (has_table_ids || num_tables > 1);                    \
        DataType value_dtype = ctx->Attr<DataType>("dtype");                                      \
        const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");                      \
        const int64_t line_size = ctx->Attr<int64_t>("line_size");                                \
        bool need_embeddings =                                                                    \
            (line_size != embedding_size) || (value_dtype != embeddings.data_type());             \
        FusedEmbeddingTmpBufferManager<OF_PP_PAIR_FIRST(k_dtype_pair), uint8_t, uint32_t>         \
            buffer_manager(nullptr, ids.shape().elem_cnt(), ctx->parallel_ctx().parallel_num(),   \
                           need_process_table_ids, line_size, embedding_size, true,               \
                           need_embeddings, value_dtype, embeddings.data_type());                 \
        return buffer_manager.TotalBufferSize();                                                  \
      });

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_ONE_EMBEDDING_FUSED_LOOKUP_KERNEL, ID_DATA_TYPE_SEQ,
                                 FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ, EMBEDDING_DATA_TYPE_SEQ)

template<typename IDX>
class OneEmbeddingFusedLookupLocalKernelState final : public user_op::OpKernelState {
 public:
  explicit OneEmbeddingFusedLookupLocalKernelState(user_op::KernelInitContext* ctx)
      : device_index_(-1) {
    OF_CUDA_CHECK(hipGetDevice(&device_index_));
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    OF_CUDA_CHECK(hipHostMalloc(&host_num_keys_, sizeof(IDX)));
    const std::string& embedding_name = ctx->Attr<std::string>("embedding_name");
    key_value_store_ = Singleton<embedding::EmbeddingManager>::Get()->GetKeyValueStore(
        embedding_name, parallel_id);
    uint32_t max_query_length = ctx->TensorDesc4ArgNameAndIndex("ids", 0)->shape().elem_cnt();
    key_value_store_->ReserveQueryLength(max_query_length);

    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    // Note(guoran): This op have no optimizer info, so set embedding states initializer constant
    // 0, which may make error in optimizer with initial_accumulator_value like adagrad and ftrl.
    std::string state_initializer;
    MakeConstantInitializerAttr(embedding_size, line_size, {}, &state_initializer);

    std::vector<EmbeddingInitializer> initializer_param;
    std::vector<int8_t> initializer_index;
    ParseInitializers(line_size, embedding_size, state_initializer,
                      ctx->Attr<std::string>("embedding_tables"), &initializer_param,
                      &initializer_index);

    const size_t param_size_bytes = initializer_param.size() * sizeof(EmbeddingInitializer);
    OF_CUDA_CHECK(hipHostMalloc(&host_initializer_param_, param_size_bytes));
    std::memcpy(host_initializer_param_, initializer_param.data(), param_size_bytes);
    OF_CUDA_CHECK(hipMalloc(&device_initializer_param_, param_size_bytes));
    OF_CUDA_CHECK(hipMemcpyAsync(device_initializer_param_, host_initializer_param_,
                                  param_size_bytes, hipMemcpyDefault,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));

    const size_t index_size_bytes = initializer_index.size() * sizeof(int8_t);
    OF_CUDA_CHECK(hipHostMalloc(&host_initializer_index_, index_size_bytes));
    std::memcpy(host_initializer_index_, initializer_index.data(), index_size_bytes);
    OF_CUDA_CHECK(hipMalloc(&device_initializer_index_, index_size_bytes));
    OF_CUDA_CHECK(hipMemcpyAsync(device_initializer_index_, host_initializer_index_,
                                  index_size_bytes, hipMemcpyDefault,
                                  ctx->stream()->As<ep::CudaStream>()->hip_stream()));
  }
  ~OneEmbeddingFusedLookupLocalKernelState() override {
    CudaCurrentDeviceGuard guard(device_index_);
    OF_CUDA_CHECK(hipHostFree(host_num_keys_));
    OF_CUDA_CHECK(hipHostFree(host_initializer_param_));
    OF_CUDA_CHECK(hipFree(device_initializer_param_));
    OF_CUDA_CHECK(hipHostFree(host_initializer_index_));
    OF_CUDA_CHECK(hipFree(device_initializer_index_));
  }

  IDX* HostNumKeys() { return host_num_keys_; }

  embedding::KeyValueStore* KeyValueStore() { return key_value_store_; }

  const int8_t* InitializerIndex() { return device_initializer_index_; }
  const EmbeddingInitializer* Initializers() { return device_initializer_param_; }

 private:
  int device_index_;
  IDX* host_num_keys_;
  embedding::KeyValueStore* key_value_store_;

  EmbeddingInitializer* host_initializer_param_;
  EmbeddingInitializer* device_initializer_param_;
  int8_t* host_initializer_index_;
  int8_t* device_initializer_index_;
};

template<typename T, typename K, typename U, typename IDX>
void LookupAndInitMissing(ep::Stream* stream,
                          OneEmbeddingFusedLookupLocalKernelState<IDX>* kernel_state, uint64_t seed,
                          uint32_t num_unique, const int64_t embedding_size,
                          const int64_t line_size, const bool put_to_store, const void* unique_ids,
                          const void* table_ids, void* num_missing_ptr, void* missing_indices,
                          void* store_values) {
  embedding::KeyValueStore* store = kernel_state->KeyValueStore();
  const EmbeddingInitializer* initializer_param = kernel_state->Initializers();
  const int8_t* initializer_index = kernel_state->InitializerIndex();
  void* host_num_keys = kernel_state->HostNumKeys();
  LookupAndInitMissing<T, K, U, IDX>(stream, seed, store, initializer_param, initializer_index,
                                     host_num_keys, num_unique, embedding_size, line_size,
                                     put_to_store, unique_ids, table_ids, num_missing_ptr,
                                     missing_indices, store_values);
}

template<typename K, typename U, typename IDX>
class FusedLocalEmbeddingTmpBufferManager final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(FusedLocalEmbeddingTmpBufferManager);
  FusedLocalEmbeddingTmpBufferManager(void* ptr, const int64_t num_ids, bool need_process_table_ids,
                                      int64_t line_size, int64_t embedding_size,
                                      bool need_embeddings, DataType value_dtype,
                                      DataType embedding_dtype)
      : offset_(0),
        offsets_(static_cast<size_t>(FusedEmbeddingBufferType::kMaxType), -1),
        sizes_(static_cast<size_t>(FusedEmbeddingBufferType::kMaxType)),
        ptr_(ptr) {
    // id shuffle
    const size_t table_ids_bytes = need_process_table_ids ? num_ids * sizeof(U) : 0;
    AllocBuffer(FusedEmbeddingBufferType::kTableIds, table_ids_bytes);
    const size_t hash_table_capacity = num_ids;
    AllocBuffer(FusedEmbeddingBufferType::kWorkspace,
                hash_table_capacity * sizeof(data_shuffle::TableEntry<K>));
    size_t cur_rank_num_ids = num_ids;
    size_t cur_rank_num_table_ids = cur_rank_num_ids;
    size_t cur_rank_num_unique_bytes = sizeof(uint32_t);
    AllocBuffer(FusedEmbeddingBufferType::kCurRankNumUnique, cur_rank_num_unique_bytes);
    size_t cur_rank_unique_ids_bytes = cur_rank_num_ids * sizeof(K);
    AllocBuffer(FusedEmbeddingBufferType::kCurRankUniqueIds, cur_rank_unique_ids_bytes);
    size_t cur_rank_unique_table_ids_bytes = cur_rank_num_table_ids * sizeof(U);
    AllocBuffer(FusedEmbeddingBufferType::kCurRankUniqueTableIds, cur_rank_unique_table_ids_bytes);
    size_t cur_rank_inverse_indices_bytes = cur_rank_num_ids * sizeof(IDX);
    AllocBuffer(FusedEmbeddingBufferType::kCurRankInverseIndices, cur_rank_inverse_indices_bytes);
    // embedding lookup
    size_t num_missing_bytes = sizeof(uint32_t);
    AllocBuffer(FusedEmbeddingBufferType::kNumMissing, num_missing_bytes);
    size_t missing_indices_bytes = cur_rank_num_ids * sizeof(uint32_t);
    AllocBuffer(FusedEmbeddingBufferType::kMissingIndices, missing_indices_bytes);
    size_t cur_rank_unique_values_bytes =
        cur_rank_num_ids * line_size * GetSizeOfDataType(value_dtype);
    AllocBuffer(FusedEmbeddingBufferType::kCurRankUniqueValues, cur_rank_unique_values_bytes);
    if (need_embeddings) {
      size_t cur_rank_unique_embeddings_bytes =
          cur_rank_num_ids * embedding_size * GetSizeOfDataType(embedding_dtype);
      AllocBuffer(FusedEmbeddingBufferType::kCurRankUniqueEmbeddings,
                  cur_rank_unique_embeddings_bytes);
    }
  }

  template<typename T = void>
  T* Ptr(FusedEmbeddingBufferType type) const {
    CHECK(ptr_ != nullptr);
    int64_t offset = offsets_.at(static_cast<size_t>(type));
    CHECK_NE(offset, -1);
    return reinterpret_cast<T*>(reinterpret_cast<char*>(ptr_) + offset);
  }

  int64_t Size(FusedEmbeddingBufferType type) const { return sizes_.at(static_cast<size_t>(type)); }

  size_t TotalBufferSize() const { return offset_; }

 private:
  void AllocBuffer(FusedEmbeddingBufferType type, size_t size) {
    const size_t type_id = static_cast<size_t>(type);
    CHECK_EQ(offsets_.at(type_id), -1);
    offsets_.at(type_id) = offset_;
    sizes_.at(type_id) = size;
    offset_ += GetCudaAlignedSize(size);
  }
  size_t offset_;
  std::vector<int64_t> offsets_;
  std::vector<int64_t> sizes_;
  void* ptr_;
};

template<typename K, typename T, typename V, typename U, typename IDX>
class OneEmbeddingFusedLookupLocalKernel final : public user_op::OpKernel {
 public:
  OneEmbeddingFusedLookupLocalKernel() = default;
  ~OneEmbeddingFusedLookupLocalKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<OneEmbeddingFusedLookupLocalKernelState<IDX>>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    // IDX type is uint32_t, table_ids type is uint8_t.
    DataType num_unique_matrix_dtype = DataType::kUInt32;
    DataType table_ids_dtype = DataType::kUInt8;
    CHECK_EQ(sizeof(IDX), GetSizeOfDataType(num_unique_matrix_dtype));
    CHECK_EQ(sizeof(U), GetSizeOfDataType(table_ids_dtype));
    auto* kernel_state = dynamic_cast<OneEmbeddingFusedLookupLocalKernelState<IDX>*>(state);
    CHECK(kernel_state != nullptr);
    const user_op::Tensor* ids = ctx->Tensor4ArgNameAndIndex("ids", 0);
    user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
    const int32_t num_tables = ctx->Attr<int32_t>("num_tables");
    // default uint8_t as table_ids type, so num_tables can not greater than 256.
    CHECK_LE(num_tables, 256) << num_tables;
    const bool has_table_ids = ctx->has_input("table_ids", 0);
    const bool need_process_table_ids = (has_table_ids || num_tables > 1);
    const int64_t num_ids = ids->shape_view().elem_cnt();
    hipStream_t hip_stream = ctx->stream()->As<ep::CudaStream>()->hip_stream();
    DataType value_dtype = ctx->Attr<DataType>("dtype");
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const int64_t padding_idx = ctx->Attr<int64_t>("padding_idx");
    const bool has_padding_idx = ctx->Attr<bool>("has_padding_idx");
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    bool need_embeddings =
        (line_size != embedding_size) || (value_dtype != embeddings->data_type());
    FusedLocalEmbeddingTmpBufferManager<K, U, IDX> buffer_manager(
        tmp_buffer->mut_dptr(), num_ids, need_process_table_ids, line_size, embedding_size,
        need_embeddings, value_dtype, embeddings->data_type());
    CHECK_GE(tmp_buffer->shape_view().elem_cnt(), buffer_manager.TotalBufferSize());
    IDX* host_num_keys = kernel_state->HostNumKeys();

    const U* table_ids_ptr = nullptr;
    if (need_process_table_ids) {
      U* tmp_table_ids_ptr = buffer_manager.template Ptr<U>(FusedEmbeddingBufferType::kTableIds);
      table_ids_ptr = tmp_table_ids_ptr;
      if (has_table_ids) {
        // use table_id default data_type uint8, if has input table_ids with different data_type,
        // cast it to uint8.
        const user_op::Tensor* table_ids = ctx->Tensor4ArgNameAndIndex("table_ids", 0);
        if (table_ids->data_type() != table_ids_dtype) {
          std::unique_ptr<ep::primitive::Cast> cast_primitive =
              ep::primitive::NewPrimitive<ep::primitive::CastFactory>(
                  DeviceType::kCUDA, table_ids->data_type(), table_ids_dtype);
          cast_primitive->Launch(ctx->stream(), table_ids->dptr(), tmp_table_ids_ptr,
                                 table_ids->shape_view().elem_cnt());
        } else {
          table_ids_ptr = reinterpret_cast<const U*>(table_ids->dptr());
        }
      } else {
        const int32_t num_tables = ctx->Attr<int32_t>("num_tables");
        data_shuffle::GenerateTableIds<<<BlocksNum4ThreadsNum(num_ids), kCudaThreadsNumPerBlock, 0,
                                         hip_stream>>>(num_ids, num_tables, tmp_table_ids_ptr);
      }
    }
    IDX* num_unique_ptr =
        buffer_manager.template Ptr<IDX>(FusedEmbeddingBufferType::kCurRankNumUnique);
    K* unique_ids_ptr = buffer_manager.template Ptr<K>(FusedEmbeddingBufferType::kCurRankUniqueIds);
    U* unique_table_ids_ptr =
        buffer_manager.template Ptr<U>(FusedEmbeddingBufferType::kCurRankUniqueTableIds);
    IDX* inverse_indices_ptr =
        buffer_manager.template Ptr<IDX>(FusedEmbeddingBufferType::kCurRankInverseIndices);
    void* workspace_ptr = buffer_manager.Ptr(FusedEmbeddingBufferType::kWorkspace);
    const size_t workspace_bytes = buffer_manager.Size(FusedEmbeddingBufferType::kWorkspace);
    int64_t hash_capacity = num_ids;
    data_shuffle::UniqueAndPartition<K, U, IDX, embedding::GlobalUniqueHash>(
        hip_stream, num_ids, hash_capacity, 1, reinterpret_cast<const K*>(ids->dptr()),
        table_ids_ptr, num_unique_ptr, unique_ids_ptr, unique_table_ids_ptr, inverse_indices_ptr,
        reinterpret_cast<data_shuffle::TableEntry<K>*>(workspace_ptr), workspace_bytes,
        need_process_table_ids, has_padding_idx, padding_idx);

    OF_CUDA_CHECK(hipMemcpyAsync(host_num_keys, num_unique_ptr, sizeof(IDX), hipMemcpyDefault,
                                  hip_stream));
    CHECK_JUST(ctx->stream()->Sync());

    uint32_t num_unique = *host_num_keys;

    // lookup and put, if is_full_cache, not put to store.
    uint32_t* num_missing_ptr =
        buffer_manager.template Ptr<uint32_t>(FusedEmbeddingBufferType::kNumMissing);
    uint32_t* missing_indices_ptr =
        buffer_manager.template Ptr<uint32_t>(FusedEmbeddingBufferType::kMissingIndices);
    void* values_ptr =
        buffer_manager.template Ptr<V>(FusedEmbeddingBufferType::kCurRankUniqueValues);
    T* cur_rank_embeddings_ptr =
        need_embeddings
            ? buffer_manager.template Ptr<T>(FusedEmbeddingBufferType::kCurRankUniqueEmbeddings)
            : reinterpret_cast<T*>(values_ptr);
    const bool is_full_cache = ctx->Attr<bool>("is_full_cache");
    const bool put_to_store = (!is_full_cache);
    const int64_t seed = ctx->Attr<int64_t>("seed");
    LookupAndInitMissing<V, K, U, IDX>(
        ctx->stream(), kernel_state, seed, num_unique, embedding_size, line_size, put_to_store,
        unique_ids_ptr, unique_table_ids_ptr, num_missing_ptr, missing_indices_ptr, values_ptr);
    if (need_embeddings) {
      CopyValuesToEmbeddings<V>(ctx->stream(), num_unique, embedding_size, line_size, value_dtype,
                                embeddings->data_type(), reinterpret_cast<V*>(values_ptr),
                                cur_rank_embeddings_ptr);
    }
    // gather
    GatherKernelUtilImpl<DeviceType::kCUDA, T, IDX>::Forward(
        ctx->stream(), inverse_indices_ptr, num_ids, cur_rank_embeddings_ptr,
        Shape({1, num_unique, embedding_size}), embeddings->mut_dptr<T>(), 0);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

// Note(guoran): Default use U type as uint8_t, IDX as uint32_t. Because table_ids is optional, so
// can not use it in hob, if has table_ids input and dtype is not uint8_t cast to uint8_t in kernel.
#define REGISTER_CUDA_ONE_EMBEDDING_FUSED_LOOKUP_LOCAL_KERNEL(k_dtype_pair, t_dtype_pair,         \
                                                              v_dtype_pair)                       \
  REGISTER_USER_KERNEL("one_embedding_fused_lookup")                                              \
      .SetCreateFn<OneEmbeddingFusedLookupLocalKernel<                                            \
          OF_PP_PAIR_FIRST(k_dtype_pair), OF_PP_PAIR_FIRST(t_dtype_pair),                         \
          OF_PP_PAIR_FIRST(v_dtype_pair), uint8_t, uint32_t>>()                                   \
      .SetIsMatchedHob(                                                                           \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                         \
          && (user_op::HobDataType("ids", 0) == OF_PP_PAIR_SECOND(k_dtype_pair))                  \
          && (user_op::HobDataType("embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair))           \
          && (user_op::HobAttr<DataType>("dtype") == OF_PP_PAIR_SECOND(v_dtype_pair))             \
          && SingleDeviceKernel())                                                                \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                         \
        const user_op::TensorDesc& ids = ctx->InputTensorDesc("ids", 0);                          \
        const user_op::TensorDesc& embeddings = ctx->OutputTensorDesc("embeddings", 0);           \
        const bool has_table_ids = ctx->has_input("table_ids", 0);                                \
        const int32_t num_tables = ctx->Attr<int32_t>("num_tables");                              \
        const bool need_process_table_ids = (has_table_ids || num_tables > 1);                    \
        DataType value_dtype = ctx->Attr<DataType>("dtype");                                      \
        const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");                      \
        const int64_t line_size = ctx->Attr<int64_t>("line_size");                                \
        bool need_embeddings =                                                                    \
            (line_size != embedding_size) || (value_dtype != embeddings.data_type());             \
        FusedLocalEmbeddingTmpBufferManager<OF_PP_PAIR_FIRST(k_dtype_pair), uint8_t, uint32_t>    \
            buffer_manager(nullptr, ids.shape().elem_cnt(), need_process_table_ids, line_size,    \
                           embedding_size, need_embeddings, value_dtype, embeddings.data_type()); \
        return buffer_manager.TotalBufferSize();                                                  \
      });

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_ONE_EMBEDDING_FUSED_LOOKUP_LOCAL_KERNEL,
                                 ID_DATA_TYPE_SEQ, FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ,
                                 EMBEDDING_DATA_TYPE_SEQ)

class OneEmbeddingFusedLookupGradKernel final : public user_op::OpKernel {
 public:
  OneEmbeddingFusedLookupGradKernel() = default;
  ~OneEmbeddingFusedLookupGradKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    // do nothing
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("one_embedding_fused_lookup_grad")
    .SetCreateFn<OneEmbeddingFusedLookupGradKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA));

}  // namespace oneflow
