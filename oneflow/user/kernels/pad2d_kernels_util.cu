/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <cstdint>
#ifdef WITH_CUDA
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/pad2d_kernels_util.h"

namespace oneflow {
namespace user_op {

template<typename IN_T>
__global__ void DoCUDAReflectionPad2d(const IN_T* src, IN_T* dest,
                                      const NdIndexOffsetHelper<int64_t, 4> index_helper,
                                      int64_t elem_num, int64_t src_num, int64_t dest_num,
                                      int64_t y_height, int64_t y_width, int64_t x_height,
                                      int64_t x_width, int64_t pad_left, int64_t pad_top) {
  DoReflectionPad2d<IN_T>(src, dest, index_helper, elem_num, src_num, dest_num, y_height, y_width,
                          x_height, x_width, pad_left, pad_top);
};

template<typename IN_T>
__global__ void DoCUDAReflectionPad2dGrad(const IN_T* src, IN_T* dest,
                                          const NdIndexOffsetHelper<int64_t, 4> index_helper,
                                          int64_t elem_num, int64_t src_num, int64_t dest_num,
                                          int64_t dy_height, int64_t dy_width, int64_t dx_height,
                                          int64_t dx_width, int64_t pad_left, int64_t pad_top) {
  DoReflectionPad2dGrad<IN_T>(src, dest, index_helper, elem_num, src_num, dest_num, dy_height,
                              dy_width, dx_height, dx_width, pad_left, pad_top);
};

template<typename IN_T>
struct ReflectionPad2dFunctor<DeviceType::kGPU, IN_T> final {
  void operator()(DeviceCtx* ctx, const IN_T* src, IN_T* dest,
                  const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch,
                  int64_t n_channel, int64_t y_height, int64_t y_width, int64_t x_height,
                  int64_t x_width, int64_t pad_left, int64_t pad_top) {
    int64_t dest_num = n_channel * y_height * y_width;
    int64_t src_num = n_channel * x_height * x_width;
    int64_t elem_num = n_batch * dest_num;
    DoCUDAReflectionPad2d<IN_T>
        <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
            src, dest, index_helper, elem_num, src_num, dest_num, y_height, y_width, x_height,
            x_width, pad_left, pad_top);
  }
};

// float16 implementation
template<>
void ReflectionPad2dFunctor<DeviceType::kGPU, float16>::operator()(
    DeviceCtx* ctx, const float16* src, float16* dest,
    const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch, int64_t n_channel,
    int64_t y_height, int64_t y_width, int64_t x_height, int64_t x_width, int64_t pad_left,
    int64_t pad_top) {
  int64_t dest_num = n_channel * y_height * y_width;
  int64_t src_num = n_channel * x_height * x_width;
  int64_t elem_num = n_batch * dest_num;
  DoCUDAReflectionPad2d<half>
      <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
          reinterpret_cast<const half*>(src), reinterpret_cast<half*>(dest), index_helper, elem_num,
          src_num, dest_num, y_height, y_width, x_height, x_width, pad_left, pad_top);
}

template<typename IN_T>
struct ReflectionPad2dGradFunctor<DeviceType::kGPU, IN_T> final {
  void operator()(DeviceCtx* ctx, const IN_T* src, IN_T* dest,
                  const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch,
                  int64_t n_channel, int64_t dy_height, int64_t dy_width, int64_t dx_height,
                  int64_t dx_width, int64_t pad_left, int64_t pad_top) {
    int64_t dest_num = n_channel * dx_height * dx_width;
    int64_t src_num = n_channel * dy_height * dy_width;
    int64_t elem_num = n_batch * src_num;
    DoCUDAReflectionPad2dGrad<IN_T>
        <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
            src, dest, index_helper, elem_num, src_num, dest_num, dy_height, dy_width, dx_height,
            dx_width, pad_left, pad_top);
  }
};

// float16 implementation
template<>
void ReflectionPad2dGradFunctor<DeviceType::kGPU, float16>::operator()(
    DeviceCtx* ctx, const float16* src, float16* dest,
    const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch, int64_t n_channel,
    int64_t dy_height, int64_t dy_width, int64_t dx_height, int64_t dx_width, int64_t pad_left,
    int64_t pad_top) {
  int64_t dest_num = n_channel * dx_height * dx_width;
  int64_t src_num = n_channel * dy_height * dy_width;
  int64_t elem_num = n_batch * src_num;
  DoCUDAReflectionPad2dGrad<half>
      <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
          reinterpret_cast<const half*>(src), reinterpret_cast<half*>(dest), index_helper, elem_num,
          src_num, dest_num, dy_height, dy_width, dx_height, dx_width, pad_left, pad_top);
}

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_REFLECTION_PAD2D_FUNCTOR,
                                 OF_PP_MAKE_TUPLE_SEQ(DeviceType::kGPU), PADDING_DATA_TYPE_GPU_SEQ);

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_REFLECTION_PAD2D_GRAD_FUNCTOR,
                                 OF_PP_MAKE_TUPLE_SEQ(DeviceType::kGPU), PADDING_DATA_TYPE_GPU_SEQ);

template<typename IN_T>
__global__ void DoCUDAReplicationPad2d(const IN_T* src, IN_T* dest,
                                       const NdIndexOffsetHelper<int64_t, 4> index_helper,
                                       int64_t elem_num, int64_t src_num, int64_t dest_num,
                                       int64_t y_height, int64_t y_width, int64_t x_height,
                                       int64_t x_width, int64_t pad_left, int64_t pad_top) {
  DoReplicationPad2d<IN_T>(src, dest, index_helper, elem_num, src_num, dest_num, y_height, y_width,
                           x_height, x_width, pad_left, pad_top);
};

template<typename IN_T>
__global__ void DoCUDAReplicationPad2dGrad(const IN_T* src, IN_T* dest,
                                           const NdIndexOffsetHelper<int64_t, 4> index_helper,
                                           int64_t elem_num, int64_t src_num, int64_t dest_num,
                                           int64_t dy_height, int64_t dy_width, int64_t dx_height,
                                           int64_t dx_width, int64_t pad_left, int64_t pad_top) {
  DoReplicationPad2dGrad<IN_T>(src, dest, index_helper, elem_num, src_num, dest_num, dy_height,
                               dy_width, dx_height, dx_width, pad_left, pad_top);
};

template<typename IN_T>
struct ReplicationPad2dFunctor<DeviceType::kGPU, IN_T> final {
  void operator()(DeviceCtx* ctx, const IN_T* src, IN_T* dest,
                  const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch,
                  int64_t n_channel, int64_t y_height, int64_t y_width, int64_t x_height,
                  int64_t x_width, int64_t pad_left, int64_t pad_top) {
    int64_t dest_num = n_channel * y_height * y_width;
    int64_t src_num = n_channel * x_height * x_width;
    int64_t elem_num = n_batch * dest_num;
    DoCUDAReplicationPad2d<IN_T>
        <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
            src, dest, index_helper, elem_num, src_num, dest_num, y_height, y_width, x_height,
            x_width, pad_left, pad_top);
  }
};

// float16 implementation
template<>
void ReplicationPad2dFunctor<DeviceType::kGPU, float16>::operator()(
    DeviceCtx* ctx, const float16* src, float16* dest,
    const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch, int64_t n_channel,
    int64_t y_height, int64_t y_width, int64_t x_height, int64_t x_width, int64_t pad_left,
    int64_t pad_top) {
  int64_t dest_num = n_channel * y_height * y_width;
  int64_t src_num = n_channel * x_height * x_width;
  int64_t elem_num = n_batch * dest_num;
  DoCUDAReplicationPad2d<half>
      <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
          reinterpret_cast<const half*>(src), reinterpret_cast<half*>(dest), index_helper, elem_num,
          src_num, dest_num, y_height, y_width, x_height, x_width, pad_left, pad_top);
}

template<typename IN_T>
struct ReplicationPad2dGradFunctor<DeviceType::kGPU, IN_T> final {
  void operator()(DeviceCtx* ctx, const IN_T* src, IN_T* dest,
                  const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch,
                  int64_t n_channel, int64_t dy_height, int64_t dy_width, int64_t dx_height,
                  int64_t dx_width, int64_t pad_left, int64_t pad_top) {
    int64_t dest_num = n_channel * dx_height * dx_width;
    int64_t src_num = n_channel * dy_height * dy_width;
    int64_t elem_num = n_batch * src_num;
    DoCUDAReplicationPad2dGrad<IN_T>
        <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
            src, dest, index_helper, elem_num, src_num, dest_num, dy_height, dy_width, dx_height,
            dx_width, pad_left, pad_top);
  }
};

// float16 implementation
template<>
void ReplicationPad2dGradFunctor<DeviceType::kGPU, float16>::operator()(
    DeviceCtx* ctx, const float16* src, float16* dest,
    const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch, int64_t n_channel,
    int64_t dy_height, int64_t dy_width, int64_t dx_height, int64_t dx_width, int64_t pad_left,
    int64_t pad_top) {
  int64_t dest_num = n_channel * dx_height * dx_width;
  int64_t src_num = n_channel * dy_height * dy_width;
  int64_t elem_num = n_batch * src_num;
  DoCUDAReplicationPad2dGrad<half>
      <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
          reinterpret_cast<const half*>(src), reinterpret_cast<half*>(dest), index_helper, elem_num,
          src_num, dest_num, dy_height, dy_width, dx_height, dx_width, pad_left, pad_top);
}

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_REPLICATION_PAD2D_FUNCTOR,
                                 OF_PP_MAKE_TUPLE_SEQ(DeviceType::kGPU), PADDING_DATA_TYPE_GPU_SEQ);

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_REPLICATION_PAD2D_GRAD_FUNCTOR,
                                 OF_PP_MAKE_TUPLE_SEQ(DeviceType::kGPU), PADDING_DATA_TYPE_GPU_SEQ);

template<typename IN_T>
__global__ void DoCUDAConstantPad2d(const IN_T* src, IN_T* dest,
                                    const NdIndexOffsetHelper<int64_t, 4> index_helper,
                                    int64_t elem_num, int64_t src_num, int64_t dest_num,
                                    int64_t y_height, int64_t y_width, int64_t x_height,
                                    int64_t x_width, int64_t pad_left, int64_t pad_top,
                                    const IN_T const_value) {
  DoConstantPad2d<IN_T>(src, dest, index_helper, elem_num, src_num, dest_num, y_height, y_width,
                        x_height, x_width, pad_left, pad_top, const_value);
};

template<typename IN_T>
__global__ void DoCUDAConstantPad2dGrad(const IN_T* src, IN_T* dest,
                                        const NdIndexOffsetHelper<int64_t, 4> index_helper,
                                        int64_t elem_num, int64_t src_num, int64_t dest_num,
                                        int64_t dy_height, int64_t dy_width, int64_t dx_height,
                                        int64_t dx_width, int64_t pad_left, int64_t pad_top) {
  DoConstantPad2dGrad<IN_T>(src, dest, index_helper, elem_num, src_num, dest_num, dy_height,
                            dy_width, dx_height, dx_width, pad_left, pad_top);
};

template<typename IN_T>
struct ConstantPad2dFunctor<DeviceType::kGPU, IN_T> final {
  void operator()(DeviceCtx* ctx, const IN_T* src, IN_T* dest,
                  const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch,
                  int64_t n_channel, int64_t y_height, int64_t y_width, int64_t x_height,
                  int64_t x_width, int64_t pad_left, int64_t pad_top, const IN_T const_value) {
    int64_t dest_num = n_channel * y_height * y_width;
    int64_t src_num = n_channel * x_height * x_width;
    int64_t elem_num = n_batch * dest_num;
    DoCUDAConstantPad2d<IN_T>
        <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
            src, dest, index_helper, elem_num, src_num, dest_num, y_height, y_width, x_height,
            x_width, pad_left, pad_top, const_value);
  }
};

// float16 implementation
template<>
void ConstantPad2dFunctor<DeviceType::kGPU, float16>::operator()(
    DeviceCtx* ctx, const float16* src, float16* dest,
    const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch, int64_t n_channel,
    int64_t y_height, int64_t y_width, int64_t x_height, int64_t x_width, int64_t pad_left,
    int64_t pad_top, const float16 const_value) {
  int64_t dest_num = n_channel * y_height * y_width;
  int64_t src_num = n_channel * x_height * x_width;
  int64_t elem_num = n_batch * dest_num;
  DoCUDAConstantPad2d<half>
      <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
          reinterpret_cast<const half*>(src), reinterpret_cast<half*>(dest), index_helper, elem_num,
          src_num, dest_num, y_height, y_width, x_height, x_width, pad_left, pad_top,
          static_cast<const half>(const_value));
}

template<typename IN_T>
struct ConstantPad2dGradFunctor<DeviceType::kGPU, IN_T> final {
  void operator()(DeviceCtx* ctx, const IN_T* src, IN_T* dest,
                  const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch,
                  int64_t n_channel, int64_t dy_height, int64_t dy_width, int64_t dx_height,
                  int64_t dx_width, int64_t pad_left, int64_t pad_top) {
    int64_t dest_num = n_channel * dx_height * dx_width;
    int64_t src_num = n_channel * dy_height * dy_width;
    int64_t elem_num = n_batch * src_num;
    DoCUDAConstantPad2dGrad<IN_T>
        <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
            src, dest, index_helper, elem_num, src_num, dest_num, dy_height, dy_width, dx_height,
            dx_width, pad_left, pad_top);
  }
};

// float16 implementation
template<>
void ConstantPad2dGradFunctor<DeviceType::kGPU, float16>::operator()(
    DeviceCtx* ctx, const float16* src, float16* dest,
    const NdIndexOffsetHelper<int64_t, 4>& index_helper, int64_t n_batch, int64_t n_channel,
    int64_t dy_height, int64_t dy_width, int64_t dx_height, int64_t dx_width, int64_t pad_left,
    int64_t pad_top) {
  int64_t dest_num = n_channel * dx_height * dx_width;
  int64_t src_num = n_channel * dy_height * dy_width;
  int64_t elem_num = n_batch * src_num;
  DoCUDAConstantPad2dGrad<half>
      <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
          reinterpret_cast<const half*>(src), reinterpret_cast<half*>(dest), index_helper, elem_num,
          src_num, dest_num, dy_height, dy_width, dx_height, dx_width, pad_left, pad_top);
}

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_CONSTANT_PAD2D_FUNCTOR,
                                 OF_PP_MAKE_TUPLE_SEQ(DeviceType::kGPU), PADDING_DATA_TYPE_GPU_SEQ);

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_CONSTANT_PAD2D_GRAD_FUNCTOR,
                                 OF_PP_MAKE_TUPLE_SEQ(DeviceType::kGPU), PADDING_DATA_TYPE_GPU_SEQ);
}  // namespace user_op
}  // namespace oneflow

#endif  // WITH_CUDA
