/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/user/kernels/cublas_fused_mlp_util.cuh"

// same with cublas_fused_mlp_util.cuh
#if CUDA_VERSION >= 11020

namespace oneflow {

namespace {

class FusedMatmulBiasAlgoCache {
  public:
    static FusedMatmulBiasAlgoCache* CreateCache() {
      if (FusedMatmulBiasAlgoCache::cache == nullptr) {
        FusedMatmulBiasAlgoCache::cache = new FusedMatmulBiasAlgoCache();
      }
      return FusedMatmulBiasAlgoCache::cache;
    }
    ~FusedMatmulBiasAlgoCache() = default;
    const hipblasLtMatmulAlgo_t* SelectAlgo(const ep::CudaStream* hip_stream, const CublasFusedMLPKernelCache* matmul_cache, 
      CublasScalarParameter alpha, CublasScalarParameter beta, const user_op::Tensor* weight, 
      const user_op::Tensor* x, const user_op::Tensor* add_to_output, void* y_ptr) {
      auto matmul_desc = matmul_cache->operation_desc;
      auto a_desc = matmul_cache->cublas_a_desc;
      auto b_desc = matmul_cache->cublas_b_desc;
      auto c_desc = matmul_cache->cublas_c_desc;

      int64_t seed = 0;
      std::hash<int64_t> hash_fn;

      HashMatmulDesc_(matmul_desc, &seed, hash_fn);
      HashMatrixLayoutDesc_(a_desc, &seed, hash_fn);
      HashMatrixLayoutDesc_(b_desc, &seed, hash_fn);
      HashMatrixLayoutDesc_(c_desc, &seed, hash_fn);

      auto it = map_.find(seed);
      if (it != map_.end()) {
        return &(it->second.algo);
      }

      int64_t row, col;
      size_t size_to_write;
      hipblasLtMatrixLayoutGetAttribute(
          c_desc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &row, sizeof(row), &size_to_write);
      hipblasLtMatrixLayoutGetAttribute(
          c_desc, HIPBLASLT_MATRIX_LAYOUT_COLS, &col, sizeof(col), &size_to_write);

      hipblasLtMatmulPreference_t preference = nullptr;
      size_t workspace_size = hip_stream->cublas_workspace_size();
      OF_CUBLAS_CHECK(hipblasLtMatmulPreferenceCreate(&preference));
      OF_CUBLAS_CHECK(hipblasLtMatmulPreferenceSetAttribute(preference,
                                                          HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                                          &workspace_size, sizeof(workspace_size)));
      int returned_results = 0;
      hipblasLtMatmulHeuristicResult_t heuristic_result[4];
      OF_CUBLAS_CHECK(hipblasLtMatmulAlgoGetHeuristic(
          hip_stream->cublas_lt_handle(), matmul_cache->operation_desc, matmul_cache->cublas_a_desc,
          matmul_cache->cublas_b_desc, matmul_cache->cublas_c_desc, matmul_cache->cublas_c_desc,
          preference, 4, heuristic_result, &returned_results)); //TODO: magic number 4
      CHECK_GT(returned_results, 0);
      hipblasLtMatmulPreferenceDestroy(preference);    

      hipEvent_t st, ed;
      float ms;
      hipEventCreate(&st);
      hipEventCreate(&ed);
      std::vector<std::pair<int, float>> sorted_algos;

      for (int i = 0; i < returned_results; i++) {
        for (int j = 0; j < 128; j++) { //TODO: magic number 128
          OF_CUBLAS_CHECK(hipblasLtMatmul(
          hip_stream->cublas_lt_handle(), matmul_cache->operation_desc, &alpha, weight->dptr(),
          matmul_cache->cublas_a_desc, x->dptr(), matmul_cache->cublas_b_desc, &beta,
          (add_to_output == nullptr) ? y_ptr : add_to_output->dptr(), matmul_cache->cublas_c_desc,
          y_ptr, matmul_cache->cublas_c_desc, &heuristic_result[i].algo, hip_stream->cublas_workspace(),
          hip_stream->cublas_workspace_size(), hip_stream->hip_stream()));
        }

        hipEventRecord(st);
        for (int j = 0; j < 128; j++) { //TODO: magic number 128
          OF_CUBLAS_CHECK(hipblasLtMatmul(
          hip_stream->cublas_lt_handle(), matmul_cache->operation_desc, &alpha, weight->dptr(),
          matmul_cache->cublas_a_desc, x->dptr(), matmul_cache->cublas_b_desc, &beta,
          (add_to_output == nullptr) ? y_ptr : add_to_output->dptr(), matmul_cache->cublas_c_desc,
          y_ptr, matmul_cache->cublas_c_desc, &heuristic_result[i].algo, hip_stream->cublas_workspace(),
          hip_stream->cublas_workspace_size(), hip_stream->hip_stream()));
        }
        hipEventRecord(ed);
        hipEventSynchronize(ed);
        hipEventElapsedTime(&ms, st, ed);

        sorted_algos.push_back(std::pair<int, float>(i, ms));
      }

      std::sort(sorted_algos.begin(), sorted_algos.end(), [](auto pair1, auto pair2) {return pair1.second < pair2.second;});

      int fastest_result_id = sorted_algos[0].first;
      map_[seed] = heuristic_result[fastest_result_id];

      return &(map_[seed].algo);
    }
  
  private:
    void HashMatmulDesc_(hipblasLtMatmulDesc_t desc,
                       int64_t* seed,
                       const std::hash<int64_t>& hash_fn) {
      size_t size_to_write;
      int trans_a, trans_b;
      uint32_t epilogue;

      hipblasLtMatmulDescGetAttribute(desc,
                                      HIPBLASLT_MATMUL_DESC_TRANSA,
                                      &trans_a,
                                      sizeof(trans_a),
                                      &size_to_write);
      HashValue_(seed, hash_fn, static_cast<int64_t>(trans_a));

      hipblasLtMatmulDescGetAttribute(desc,
                                      HIPBLASLT_MATMUL_DESC_TRANSB,
                                      &trans_b,
                                      sizeof(trans_b),
                                      &size_to_write);
      HashValue_(seed, hash_fn, static_cast<int64_t>(trans_b));

      hipblasLtMatmulDescGetAttribute(desc,
                                      HIPBLASLT_MATMUL_DESC_EPILOGUE,
                                      &epilogue,
                                      sizeof(epilogue),
                                      &size_to_write);
      HashValue_(seed, hash_fn, static_cast<int64_t>(epilogue));
    }

      void HashMatrixLayoutDesc_(hipblasLtMatrixLayout_t desc,
                             int64_t* seed,
                             const std::hash<int64_t>& hash_fn) {
      size_t size_to_write;
      uint32_t dtype;
      int32_t batch;
      uint64_t row, col;
      int64_t ld, batch_offset;

      hipblasLtMatrixLayoutGetAttribute(desc,
                                        HIPBLASLT_MATRIX_LAYOUT_TYPE,
                                        &dtype,
                                        sizeof(dtype),
                                        &size_to_write);
      HashValue_(seed, hash_fn, static_cast<int64_t>(dtype));

      hipblasLtMatrixLayoutGetAttribute(
          desc,
          HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT,
          &batch,
          sizeof(batch),
          &size_to_write);
      HashValue_(seed, hash_fn, static_cast<int64_t>(batch));

      hipblasLtMatrixLayoutGetAttribute(
          desc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &row, sizeof(row), &size_to_write);
      HashValue_(seed, hash_fn, static_cast<int64_t>(row));

      hipblasLtMatrixLayoutGetAttribute(
          desc, HIPBLASLT_MATRIX_LAYOUT_COLS, &col, sizeof(col), &size_to_write);
      HashValue_(seed, hash_fn, static_cast<int64_t>(col));

      hipblasLtMatrixLayoutGetAttribute(
          desc, HIPBLASLT_MATRIX_LAYOUT_LD, &ld, sizeof(ld), &size_to_write);
      HashValue_(seed, hash_fn, static_cast<int64_t>(ld));

      hipblasLtMatrixLayoutGetAttribute(
          desc,
          HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET,
          &batch_offset,
          sizeof(batch_offset),
          &size_to_write);
      HashValue_(seed, hash_fn, static_cast<int64_t>(batch_offset));
    }

    void HashValue_(int64_t* seed,
                    const std::hash<int64_t>& hash_fn,
                    int64_t value) {
      *seed ^= hash_fn(value) + 0x9e3779b9 + (*seed << 6) + (*seed >> 2);
    }

    FusedMatmulBiasAlgoCache() = default;
    std::map<int64_t, hipblasLtMatmulHeuristicResult_t> map_;
    inline static FusedMatmulBiasAlgoCache* cache = nullptr;
};

class FusedMatmulBiasKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  FusedMatmulBiasKernel() = default;
  ~FusedMatmulBiasKernel() override = default;

  std::shared_ptr<user_op::OpKernelCache> InitOpKernelCache(
      user_op::KernelCacheContext* ctx) const override {
    return CreateCublasFusedMLPKernelCache();
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState*,
               const user_op::OpKernelCache* cache) const override {
    auto* hip_stream = ctx->stream()->As<ep::CudaStream>();
    const auto* matmul_cache = CHECK_NOTNULL(dynamic_cast<const CublasFusedMLPKernelCache*>(cache));

    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const user_op::Tensor* _add_to_output = (ctx->has_input("_add_to_output", 0))
                                                ? ctx->Tensor4ArgNameAndIndex("_add_to_output", 0)
                                                : nullptr;

    const DataType data_type = out->data_type();
    const hipblasComputeType_t cublas_compute_dtype = GetComputeType(data_type);
    const hipDataType cuda_data_type = GetCudaDataType(data_type);
    size_t cublas_m = 0, cublas_n = 0, cublas_k = 0;
    int64_t cublas_lda = 0, cublas_ldb = 0, cublas_ldc = 0;

    const double alpha = ctx->Attr<double>("alpha");
    const double beta = (ctx->has_input("_add_to_output", 0)) ? ctx->Attr<double>("beta") : 0.0;

    const auto sp_alpha = GetCublasScalarParameter(alpha, cublas_compute_dtype);
    const auto sp_beta = GetCublasScalarParameter(beta, cublas_compute_dtype);

    DimVector in_shape({x->shape_view().Count(0, x->shape_view().NumAxes() - 1),
                        x->shape_view().At(x->shape_view().NumAxes() - 1)});

    DimVector weight_shape(2);

    const user_op::Tensor* weight = ctx->Tensor4ArgNameAndIndex("weight", 0);
    const user_op::Tensor* bias = ctx->Tensor4ArgNameAndIndex("bias", 0);

    weight->shape_view().ToDimVector(&weight_shape);

    InferMatmulCublasMNK(in_shape, weight_shape,
                         /*transpose_a=*/ep::primitive::BlasTransposeType::N,
                         /*transpose_b=*/ep::primitive::BlasTransposeType::T, &cublas_m, &cublas_n,
                         &cublas_k, &cublas_lda, &cublas_ldb, &cublas_ldc);

    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_BIAS;
    void* y_ptr = ctx->Tensor4ArgNameAndIndex("out", 0)->mut_dptr();

    SetCublasAttr(matmul_cache, cublas_compute_dtype, cuda_data_type, false,
                  /*transpose_a=*/ep::primitive::BlasTransposeType::N,
                  /*transpose_b=*/ep::primitive::BlasTransposeType::T, epilogue, bias->dptr(),
                  nullptr, cublas_m, cublas_n, cublas_k, cublas_lda, cublas_ldb, cublas_ldc);
    
    FusedMatmulBiasAlgoCache* algo_cache = FusedMatmulBiasAlgoCache::CreateCache();
    const hipblasLtMatmulAlgo_t* algo = algo_cache->SelectAlgo(hip_stream, matmul_cache, sp_alpha, sp_beta, weight, x, _add_to_output, y_ptr);

    OF_CUBLAS_CHECK(hipblasLtMatmul(
        hip_stream->cublas_lt_handle(), matmul_cache->operation_desc, &sp_alpha, weight->dptr(),
        matmul_cache->cublas_a_desc, x->dptr(), matmul_cache->cublas_b_desc, &sp_beta,
        (_add_to_output == nullptr) ? y_ptr : _add_to_output->dptr(), matmul_cache->cublas_c_desc,
        y_ptr, matmul_cache->cublas_c_desc, algo, hip_stream->cublas_workspace(),
        hip_stream->cublas_workspace_size(), hip_stream->hip_stream()));
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_MATMUL_BIAS_KERNEL_GPU(data_type)               \
  REGISTER_USER_KERNEL("fused_matmul_bias")                            \
      .SetCreateFn<FusedMatmulBiasKernel>()                            \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("out", 0) == data_type));

REGISTER_FUSED_MATMUL_BIAS_KERNEL_GPU(DataType::kDouble);
REGISTER_FUSED_MATMUL_BIAS_KERNEL_GPU(DataType::kFloat);
REGISTER_FUSED_MATMUL_BIAS_KERNEL_GPU(DataType::kFloat16);
#if CUDA_VERSION >= 11000
REGISTER_FUSED_MATMUL_BIAS_KERNEL_GPU(DataType::kBFloat16);
#endif  // CUDA_VERSION >= 11000

}  // namespace

}  // namespace oneflow

#endif  // CUDA_VERSION >= 11020
