/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <cstdint>
#ifdef WITH_CUDA
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/reflection_pad2d_kernel_util.h"

namespace oneflow {
namespace user_op {


template<typename T>
__global__ void DoCUDAReflectionPad2d(
    const Tensor*  x, Tensor* y, int64_t c_idx, int64_t h_idx, int64_t w_idx, int64_t pad_left, int64_t pad_top
) {
  printf("\n .cu >>>>>>>>>>>>>>>>>>> DoCUDAReflectionPad2d !!!!!!!!!!!!!!!!!!!!!!!!!!!!!! \n");
  DoReflectionPad2d<T>(x, y, c_idx, h_idx, w_idx, pad_left, pad_top);
  printf("\n .cu >>>>>>>>>>>>>>>>>>> DoCUDAReflectionPad2d finished !!!!!!!!!!!!!!!!!!!!!!!!!!!!!! \n");
};


template<typename T>
__global__ void DoCUDAReflectionPad2dGrad(
    const Tensor*  dy, Tensor* dx, int64_t c_idx, int64_t h_idx, int64_t w_idx, int64_t pad_left, int64_t pad_top
) {
  printf("\n .cu >>>>>>>>>>>>>>>>>>> DoCUDAReflectionPad2dGrad !!!!!!!!!!!!!!!!!!!!!!!!!!!!!! \n");
  DoReflectionPad2dGrad<T>(dy, dx, c_idx, h_idx, w_idx, pad_left, pad_top);
};


template<typename T>
struct ReflectionPad2dFunctor<DeviceType::kGPU, T> final {
  void operator()(
      DeviceCtx* ctx, const Tensor*  x, Tensor* y, int64_t c_idx, int64_t h_idx, int64_t w_idx, int64_t pad_left, int64_t pad_top
    ){
    int64_t  elem_cnt = y->shape().elem_cnt();
    printf("\n.cu >>>>>>>>>>>>>>>>> ReflectionPad2dFunctor >> elem_cnt >>>>>>>>>>>>>>>>>>>>>>>>>>>%ld !\n", elem_cnt);
    RUN_CUDA_KERNEL((DoCUDAReflectionPad2d<T>), ctx, BlocksNum4ThreadsNum(elem_cnt),
                   x, y, c_idx, h_idx, w_idx, pad_left, pad_top);
  }
};


template<typename T>
struct ReflectionPad2dGradFunctor<DeviceType::kGPU, T> final {
  void operator()(
      DeviceCtx* ctx, const Tensor*  dy, Tensor* dx, int64_t c_idx, int64_t h_idx, int64_t w_idx, int64_t pad_left, int64_t pad_top
    ){
    int64_t  elem_cnt = dy->shape().elem_cnt();
    printf("\n.cu >>>>>>>>>>>>>>>>> ReflectionPad2dGradFunctor >> elem_cnt >>>>>>>>>>>>>>>>>>>>>>>>>>>%ld !\n", elem_cnt);
    RUN_CUDA_KERNEL((DoCUDAReflectionPad2dGrad<T>), ctx, BlocksNum4ThreadsNum(elem_cnt),
                   dy, dx, c_idx, h_idx, w_idx, pad_left, pad_top);
  }
};



OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_REFLECTION_PAD2D_FUNCTOR,
                                 OF_PP_MAKE_TUPLE_SEQ(DeviceType::kGPU),
                                 REFLECTION_PAD2D_DATA_TYPE_GPU_SEQ);

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_REFLECTION_PAD2D_GRAD_FUNCTOR,
                                 OF_PP_MAKE_TUPLE_SEQ(DeviceType::kGPU),
                                 REFLECTION_PAD2D_GRAD_DATA_TYPE_GPU_SEQ);


}  // namespace user_op
}  // namespace oneflow

#endif  // WITH_CUDA