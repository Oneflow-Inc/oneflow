/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/cuda/softmax.cuh"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/user/kernels/fused_softmax.cuh"

namespace oneflow {

namespace {

template<typename T, typename ComputeType, typename MASK, size_t num_dims>
void LaunchBroadcastForwardKernel(hipStream_t stream, const T* x, T* y, const MASK* mask,
                                  const int64_t elem_cnt, const int64_t rows, const int64_t cols,
                                  const float fill, const float scale, const int64_t* input_dims,
                                  const int64_t* mask_dims) {
  NdIndexOffsetHelper<int32_t, num_dims> input_index_helper(input_dims);
  NdIndexOffsetHelper<int32_t, num_dims> mask_index_helper(mask_dims);
  cuda::fused_softmax::BroadcastMaskSoftmaxParams<num_dims, int32_t> params;
  params.src_index_helper = input_index_helper;
  params.mask_index_helper = mask_index_helper;
  params.mask_dims = mask_dims;
  params.row_size = cols;
  params.fill = fill;
  params.scale = scale;
  cuda::fused_softmax::BroadcastScaleMaskLoad<T, ComputeType, MASK, num_dims, int32_t> load(x, mask,
                                                                                            params);
  cuda::softmax::DirectStore<ComputeType, T> store(y, cols);
  OF_CUDA_CHECK((cuda::softmax::DispatchSoftmax<decltype(load), decltype(store), ComputeType>(
      stream, load, store, rows, cols)));
}

template<typename T, typename ComputeType, typename MASK>
void LaunchElementwiseForwardKernel(hipStream_t stream, const T* x, T* y, const MASK* mask,
                                    const int64_t rows, const int64_t cols, const float fill,
                                    const float scale) {
  cuda::fused_softmax::ElementwiseMaskSoftmaxParams params;
  params.row_size = cols;
  params.fill = fill;
  params.scale = scale;
  cuda::fused_softmax::ElementwiseScaleMaskLoad<T, ComputeType, MASK> load(x, mask, params);
  cuda::softmax::DirectStore<ComputeType, T> store(y, cols);
  OF_CUDA_CHECK((cuda::softmax::DispatchSoftmax<decltype(load), decltype(store), ComputeType>(
      stream, load, store, rows, cols)));
}

template<typename T, typename ComputeType, typename MASK, size_t num_dims>
void LaunchBroadcastBackwardKernel(hipStream_t stream, const T* y, const T* dy, T* dx,
                                   const MASK* mask, const int64_t elem_cnt, const int64_t rows,
                                   const int64_t cols, const float fill, const float scale,
                                   const int64_t* input_dims, const int64_t* mask_dims) {
  NdIndexOffsetHelper<int32_t, num_dims> input_index_helper(input_dims);
  NdIndexOffsetHelper<int32_t, num_dims> mask_index_helper(mask_dims);
  cuda::fused_softmax::BroadcastMaskSoftmaxParams<num_dims, int32_t> params;
  params.src_index_helper = input_index_helper;
  params.mask_index_helper = mask_index_helper;
  params.mask_dims = mask_dims;
  params.row_size = cols;
  params.fill = fill;
  params.scale = scale;
  cuda::softmax::DirectLoad<T, ComputeType> load_y(y, cols);
  cuda::softmax::DirectLoad<T, ComputeType> load_dy(dy, cols);
  cuda::fused_softmax::BroadcastScaleMaskStore<ComputeType, T, MASK, num_dims, int32_t> store(
      dx, mask, params);
  OF_CUDA_CHECK((
      cuda::softmax::DispatchSoftmaxGrad<decltype(load_y), decltype(load_dy), decltype(store),
                                         ComputeType>(stream, load_y, load_dy, store, rows, cols)));
}

template<typename T, typename ComputeType, typename MASK>
void LaunchElementwiseBackwardKernel(hipStream_t stream, const T* y, const T* dy, T* dx,
                                     const MASK* mask, const int64_t rows, const int64_t cols,
                                     const float fill, const float scale) {
  cuda::fused_softmax::ElementwiseMaskSoftmaxParams params;
  params.row_size = cols;
  params.fill = fill;
  params.scale = scale;
  cuda::softmax::DirectLoad<T, ComputeType> load_y(y, cols);
  cuda::softmax::DirectLoad<T, ComputeType> load_dy(dy, cols);
  cuda::fused_softmax::ElementwiseScaleMaskStore<ComputeType, T, MASK> store(dx, mask, params);
  OF_CUDA_CHECK((
      cuda::softmax::DispatchSoftmaxGrad<decltype(load_y), decltype(load_dy), decltype(store),
                                         ComputeType>(stream, load_y, load_dy, store, rows, cols)));
}

constexpr int32_t kMaxNumDims = 5;

template<typename T, typename MASK>
class FusedScaleMaskSoftmaxKernel final : public user_op::OpKernel {
 public:
  FusedScaleMaskSoftmaxKernel() = default;
  ~FusedScaleMaskSoftmaxKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* mask = ctx->Tensor4ArgNameAndIndex("mask", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const float mask_fill_value = ctx->Attr<float>("mask_fill_value");
    const float scale_value = ctx->Attr<float>("scale_value");
    const ShapeView& x_shape = x->shape_view();
    const ShapeView& mask_shape = mask->shape_view();
    CHECK_GE(x_shape.NumAxes(), 2);
    const int64_t elem_cnt = x_shape.elem_cnt();
    const int64_t cols = x_shape.At(x_shape.NumAxes() - 1);
    const int64_t rows = x_shape.Count(0, x_shape.NumAxes() - 1);
    const size_t num_input_dims = x_shape.NumAxes();
    const int64_t* input_dims = x_shape.ptr();
    const size_t num_mask_dims = mask_shape.NumAxes();
    const int64_t* mask_dims = mask_shape.ptr();
    using ComputeType = typename cuda::softmax::DefaultComputeType<T>::type;

    size_t simplified_num_dims = 0;
    int64_t simplified_input_dims[kMaxNumDims];
    int64_t simplified_mask_dims[kMaxNumDims];
    cuda::fused_softmax::SimplifyBroadcastDims(num_input_dims, input_dims, num_mask_dims, mask_dims,
                                               &simplified_num_dims, simplified_input_dims,
                                               simplified_mask_dims);
    if (simplified_num_dims == 1) {
      LaunchElementwiseForwardKernel<T, ComputeType, MASK>(
          ctx->stream()->As<ep::CudaStream>()->hip_stream(), x->dptr<T>(), y->mut_dptr<T>(),
          mask->dptr<MASK>(), rows, cols, mask_fill_value, scale_value);
    }
#define DEFINE_ONE_ELIF(dims)                                                               \
  else if (simplified_num_dims == dims) {                                                   \
    LaunchBroadcastForwardKernel<T, ComputeType, MASK, dims>(                               \
        ctx->stream()->As<ep::CudaStream>()->hip_stream(), x->dptr<T>(), y->mut_dptr<T>(), \
        mask->dptr<MASK>(), elem_cnt, rows, cols, mask_fill_value, scale_value,             \
        simplified_input_dims, simplified_mask_dims);                                       \
  }
    DEFINE_ONE_ELIF(2)
    DEFINE_ONE_ELIF(3)
    DEFINE_ONE_ELIF(4)
#undef DEFINE_ONE_ELIF
    else {
      UNIMPLEMENTED();
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T, typename MASK>
class FusedScaleMaskSoftmaxGradKernel final : public user_op::OpKernel {
 public:
  FusedScaleMaskSoftmaxGradKernel() = default;
  ~FusedScaleMaskSoftmaxGradKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* mask = ctx->Tensor4ArgNameAndIndex("mask", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const float scale_value = ctx->Attr<float>("scale_value");
    const float mask_fill_value = static_cast<float>(0.0);
    const ShapeView& dy_shape = dy->shape_view();
    const ShapeView& mask_shape = mask->shape_view();
    CHECK_GE(dy_shape.NumAxes(), 2);
    const int64_t elem_cnt = dy_shape.elem_cnt();
    const int64_t cols = dy_shape.At(dy_shape.NumAxes() - 1);
    const int64_t rows = dy_shape.Count(0, dy_shape.NumAxes() - 1);
    const int64_t* input_dims = dy_shape.ptr();
    const size_t num_input_dims = dy_shape.NumAxes();
    const int64_t* mask_dims = mask_shape.ptr();
    const size_t num_mask_dims = mask_shape.NumAxes();

    using ComputeType = typename cuda::softmax::DefaultComputeType<T>::type;

    size_t simplified_num_dims = 0;
    int64_t simplified_input_dims[kMaxNumDims];
    int64_t simplified_mask_dims[kMaxNumDims];
    cuda::fused_softmax::SimplifyBroadcastDims(num_input_dims, input_dims, num_mask_dims, mask_dims,
                                               &simplified_num_dims, simplified_input_dims,
                                               simplified_mask_dims);
    if (simplified_num_dims == 1) {
      LaunchElementwiseBackwardKernel<T, ComputeType, MASK>(
          ctx->stream()->As<ep::CudaStream>()->hip_stream(), y->dptr<T>(), dy->dptr<T>(),
          dx->mut_dptr<T>(), mask->dptr<MASK>(), rows, cols, mask_fill_value, scale_value);
    }
#define DEFINE_ONE_ELIF(dims)                                                                      \
  else if (simplified_num_dims == dims) {                                                          \
    LaunchBroadcastBackwardKernel<T, ComputeType, MASK, dims>(                                     \
        ctx->stream()->As<ep::CudaStream>()->hip_stream(), y->dptr<T>(), dy->dptr<T>(),           \
        dx->mut_dptr<T>(), mask->dptr<MASK>(), elem_cnt, rows, cols, mask_fill_value, scale_value, \
        simplified_input_dims, simplified_mask_dims);                                              \
  }
    DEFINE_ONE_ELIF(2)
    DEFINE_ONE_ELIF(3)
    DEFINE_ONE_ELIF(4)
#undef DEFINE_ONE_ELIF
    else {
      UNIMPLEMENTED();
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

}  // namespace

#define REGISTER_FUSED_SCALE_MASK_SOFTMAX_CUDA_KERNEL(dtype, mask_dtype)              \
  REGISTER_USER_KERNEL("fused_scale_mask_softmax")                                    \
      .SetCreateFn<FusedScaleMaskSoftmaxKernel<dtype, mask_dtype>>()                  \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobDataType("mask", 0) == GetDataType<mask_dtype>::value));

REGISTER_FUSED_SCALE_MASK_SOFTMAX_CUDA_KERNEL(half, bool)
REGISTER_FUSED_SCALE_MASK_SOFTMAX_CUDA_KERNEL(float, bool)
#undef REGISTER_FUSED_SCALE_MASK_SOFTMAX_CUDA_KERNEL

#define REGISTER_FUSED_SCALE_MASK_SOFTMAX_GRAD_KERNEL(dtype, mask_dtype)               \
  REGISTER_USER_KERNEL("fused_scale_mask_softmax_grad")                                \
      .SetCreateFn<FusedScaleMaskSoftmaxGradKernel<dtype, mask_dtype>>()               \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                 \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobDataType("mask", 0) == GetDataType<mask_dtype>::value));

REGISTER_FUSED_SCALE_MASK_SOFTMAX_GRAD_KERNEL(half, bool)
REGISTER_FUSED_SCALE_MASK_SOFTMAX_GRAD_KERNEL(float, bool)
#undef REGISTER_FUSED_SCALE_MASK_SOFTMAX_GRAD_KERNEL

}  // namespace oneflow
