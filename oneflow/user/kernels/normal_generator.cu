#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/normal_generator.h"
#include "oneflow/core/common/data_type.h"

namespace oneflow {

namespace {

template<typename T>
__device__ T GenNormal(hiprandState* state);

template<>
__device__ float GenNormal<float>(hiprandState* state) {
  return hiprand_normal(state);
}

template<>
__device__ double GenNormal<double>(hiprandState* state) {
  return hiprand_normal_double(state);
}

template<typename T>
__global__ void GenerateGpu(hiprandState* state, const int64_t elem_cnt, T* dptr) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState localState = state[id];
  if (id < elem_cnt) { dptr[id] = GenNormal<T>(&localState); }
  state[id] = localState;
}

}  // namespace

template<typename T>
void NormalGenerator<DeviceType::kGPU>::Generate(DeviceCtx* device_ctx, const int64_t elem_cnt,
                                                  T* dptr) {
  int32_t block_num = generator_->max_block_num();
  int32_t thread_num = generator_->max_thread_num();
  auto* curand_states = generator_->curand_states();
  GenerateGpu<T>
      <<<block_num, thread_num, 0, device_ctx->hip_stream()>>>(curand_states, elem_cnt, dptr);
}


#define INITIATE_GPU_NORMAL_GENERATOR(T, typeproto)                                    \
  template void NormalGenerator<DeviceType::kGPU>::Generate<T>(DeviceCtx * device_ctx, \
                                                                const int64_t elem_cnt, T* dptr);

OF_PP_FOR_EACH_TUPLE(INITIATE_GPU_NORMAL_GENERATOR, FLOATING_DATA_TYPE_SEQ);

}  // namespace oneflow