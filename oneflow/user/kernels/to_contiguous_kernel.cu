#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <type_traits>
#include "oneflow/core/common/device_type.pb.h"
#include "oneflow/user/kernels/to_contiguous_kernel.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/elementwise.cuh"

namespace oneflow {

namespace {

constexpr int32_t kThreadWorkSize = 4;
constexpr int32_t kNumThreads = 32 * 4;
constexpr int32_t get_min_threads_num() { return kNumThreads; }
constexpr int32_t get_block_work_size() { return kThreadWorkSize * kNumThreads; }
constexpr int32_t get_num_blocks(int64_t elem_cnt) {
  return (elem_cnt + get_block_work_size() - 1) / get_block_work_size();
}

struct StrideParam {
  int32_t stride[SHAPE_MAX_AXIS_SIZE];

  StrideParam(const int64_t* stride_vec, const size_t ndim) {
    for (size_t i = 0; i < ndim; ++i) { stride[i] = stride_vec[i]; }
  }
};

template<typename IndexType, size_t ndim>
__device__ __forceinline__ IndexType compute_index(IndexType out_offset,
                                                   const StrideParam& out_params,
                                                   const StrideParam& in_params) {
  IndexType in_offset = 0;
  IndexType remaining = out_offset;

#pragma unroll
  for (size_t i = 0; i < ndim; ++i) {
    const IndexType idx = static_cast<IndexType>(remaining / out_params.stride[i]);
    remaining -= idx * out_params.stride[i];
    in_offset += idx * in_params.stride[i];
  }
  return in_offset;
}

template<typename T, typename IndexType, size_t ndim>
__global__ void ToContiguousForwardGpuParallel(IndexType count, const StrideParam in_stride,
                                               const StrideParam out_stride, const T* in_dptr,
                                               T* out_dptr, const int32_t num_block_threads,
                                               const int32_t thread_work_size,
                                               const int32_t block_work_size) {
  IndexType remaining = count - block_work_size * blockIdx.x;
  IndexType idx = blockIdx.x;
  IndexType thread_idx = threadIdx.x;
#pragma unroll
  for (int32_t i = 0; i < thread_work_size; i++) {
    if (thread_idx >= remaining) { return; }
    IndexType out_idx = thread_idx + block_work_size * idx;
    IndexType in_idx = compute_index<IndexType, ndim>(out_idx, out_stride, in_stride);
    out_dptr[out_idx] = in_dptr[in_idx];
    thread_idx += num_block_threads;
  }
}

template<typename T, typename IndexType>
void LaunchToContiguousKernel(ep::Stream* stream, IndexType count, const size_t ndim,
                              IndexType block_size, const std::vector<int64_t>& in_stride,
                              const DimVector& out_stride, const char* in_dptr, char* out_dptr) {
  const int32_t num_blocks = get_num_blocks(count);
  constexpr int32_t num_threads = get_min_threads_num();
  constexpr int32_t block_work_size = get_block_work_size();
  StrideParam param_in_stride(in_stride.data(), ndim), param_out_stride(out_stride.data(), ndim);

  switch (ndim) {
#define TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(dim)                                             \
  case dim:                                                                                 \
    ToContiguousForwardGpuParallel<T, IndexType, dim>                                       \
        <<<num_blocks, num_threads, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(      \
            count, param_in_stride, param_out_stride, reinterpret_cast<const T*>(in_dptr),  \
            reinterpret_cast<T*>(out_dptr), num_threads, kThreadWorkSize, block_work_size); \
    break;

    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(1)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(2)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(3)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(4)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(5)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(6)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(7)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(8)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(9)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(10)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(11)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(12)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(13)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(14)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(15)
    TO_CONTIGUOUS_FORWARD_GPU_PARALLEL(16)
    default: break;
#undef TO_CONTIGUOUS_FORWARD_GPU_PARALLEL
  }
}

}  // namespace

template<typename T>
struct ToContiguousUtil<DeviceType::kCUDA, T> : ToContiguousUtilBase {
  using ToContiguousUtilBase::ToContiguousUtilBase;
  static constexpr size_t dsize = sizeof(T);
  void operator()() {
    int constant_memory_size = 0;
    const size_t ndims = contiguous_dim + 1;
    if (ndims == 0) {
      // 0-dim tensor
      OF_CUDA_CHECK(hipMemcpyAsync(out_dptr, in_dptr, block_size * dsize, hipMemcpyDeviceToDevice,
                                    stream->As<ep::CudaStream>()->hip_stream()));
    } else {
      bool is_same = true;
      for (int64_t i = contiguous_dim; i != -1; --i) {
        if (out_stride[i] != in_stride[i]) {
          is_same = false;
          break;
        }
      }
      if (is_same) {
        // if input tensor's strides equals to output's, than just copy one memory-contiguous tensor
        OF_CUDA_CHECK(hipMemcpyAsync(out_dptr, in_dptr, element_count * dsize,
                                      hipMemcpyDeviceToDevice,
                                      stream->As<ep::CudaStream>()->hip_stream()));
      } else {
        if (element_count < GetMaxVal<int32_t>()) {
          LaunchToContiguousKernel<T, int32_t>(stream, element_count, ndims, block_size, in_stride,
                                               out_stride, in_dptr, out_dptr);
        } else {
          LaunchToContiguousKernel<T, int64_t>(stream, element_count, ndims, block_size, in_stride,
                                               out_stride, in_dptr, out_dptr);
        }
      }
    }
  }
};

#define INSTANTIATE_TO_CONTIGUOUS_UTILS_FOR_CUDA(cpp_type, data_type) \
  template struct ToContiguousUtil<DeviceType::kCUDA, cpp_type>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_TO_CONTIGUOUS_UTILS_FOR_CUDA,
                     TO_CONTIGUOUS_COMMON_TYPES TO_CONTIGUOUS_CUDA_SPECIAL_TYPE)

}  // namespace oneflow
