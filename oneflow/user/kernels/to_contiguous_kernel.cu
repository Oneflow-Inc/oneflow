#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <type_traits>
#include "oneflow/core/common/device_type.pb.h"
#include "oneflow/core/graph/task_node.h"
#include "oneflow/user/kernels/to_contiguous_kernel.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/elementwise.cuh"

namespace oneflow {

namespace {

constexpr int kBlockSize = cuda::elementwise::kBlockSize;
__constant__ unsigned long int in_stride_vec[16];
__constant__ unsigned long int out_stride_vec[16];

int GetMinThreadNum(int64_t elem_cnt) { return std::min<int64_t>(elem_cnt, kBlockSize); }

int GetNumBlocks(int64_t elem_cnt) {
  int num_blocks = 0;
  OF_CUDA_CHECK(cuda::elementwise::GetNumBlocks(elem_cnt, &num_blocks));
  return num_blocks;
}

template<typename IndexType>
__device__ IndexType compute_index(IndexType out_offset, const size_t ndim) {
  IndexType in_offset = 0;
  IndexType remaining = out_offset;

#pragma unroll
  for (size_t i = 0; i < ndim; ++i) {
    const IndexType idx = remaining / out_stride_vec[i];
    remaining = remaining - idx * out_stride_vec[i];
    in_offset = in_offset + idx * in_stride_vec[i];
  }
  return in_offset;
}

template<typename T, typename IndexType>
__global__ void ToContiguousForwardGpu(IndexType count, size_t ndim, const T* in_dptr,
                                       T* out_dptr) {
  for (IndexType out_idx = blockIdx.x * blockDim.x + threadIdx.x, step = blockDim.x * gridDim.x;
       out_idx < count; out_idx += step) {
    IndexType in_idx = compute_index<IndexType>(out_idx, ndim);
    out_dptr[out_idx] = in_dptr[in_idx];
  }
}

template<typename T, typename IndexType, size_t pack_size>
__global__ void ToContiguousForwardGpu(IndexType count, size_t ndim, const T* in_dptr,
                                       T* out_dptr) {
  IndexType global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  for (IndexType out_idx = global_thread_id * pack_size; out_idx < count; out_idx += gridDim.x * blockDim.x * pack_size) {
    IndexType in_idx = compute_index<IndexType>(out_idx, ndim);
  #pragma unroll
    for (size_t i = 0; i < pack_size; i++) {
      out_dptr[out_idx+i] = in_dptr[in_idx+i];
    }
  }
}

template<typename T, typename IndexType>
__global__ void ToContiguousForwardGpu(IndexType count, IndexType block_size, size_t ndim, const T* in_dptr,
                                       T* out_dptr) {
  IndexType global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  for (IndexType out_idx = global_thread_id * block_size; out_idx < count; out_idx += gridDim.x * blockDim.x * block_size) {
    IndexType in_idx = compute_index<IndexType>(out_idx, ndim);
  #pragma unroll
    for (size_t i = 0; i < block_size; i++) {
      out_dptr[out_idx+i] = in_dptr[in_idx+i];
    }
  }
}


template<typename T, typename IndexType, size_t pack_size>
void LaunchToContiguousKernel(ep::Stream* stream, IndexType count, const size_t ndim,
                              IndexType block_size, const std::vector<int64_t>& in_stride,
                              const StrideVector& out_stride, const char* in_dptr, char* out_dptr) {
  const int num_blocks = GetNumBlocks(count);
  const int num_threads = GetMinThreadNum(count);
  unsigned long int tmp_in_stride[ndim]={0};
  unsigned long int tmp_out_stride[ndim]={0};
  for(size_t i=0; i < ndim; ++i){
    tmp_in_stride[i] = in_stride.at(i);
    tmp_out_stride[i] = out_stride.at(i);
  }

  OF_CUDA_CHECK(
      hipMemcpyToSymbol(HIP_SYMBOL(in_stride_vec), tmp_in_stride, ndim * sizeof(unsigned long int)));
  OF_CUDA_CHECK(
      hipMemcpyToSymbol(HIP_SYMBOL(out_stride_vec), tmp_out_stride, ndim * sizeof(unsigned long int)));


  if (pack_size == 16 && block_size % 16 == 0) {
    ToContiguousForwardGpu<T, IndexType, 16>
      <<<num_blocks, num_threads, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
          count, ndim, reinterpret_cast<const T*>(in_dptr), reinterpret_cast<T*>(out_dptr));
  } else if (pack_size == 8 && block_size % 8 == 0) {
    ToContiguousForwardGpu<T, IndexType, 8>
      <<<num_blocks, num_threads, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
          count, ndim, reinterpret_cast<const T*>(in_dptr), reinterpret_cast<T*>(out_dptr));
  } else if(pack_size == 4 && block_size % 4 == 0 ){
    ToContiguousForwardGpu<T, IndexType, 4>
      <<<num_blocks, num_threads, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
          count, ndim, reinterpret_cast<const T*>(in_dptr), reinterpret_cast<T*>(out_dptr));
  } else if(pack_size == 2 && block_size % 2 == 0 ){
    ToContiguousForwardGpu<T, IndexType, 2>
      <<<num_blocks, num_threads, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
          count, ndim, reinterpret_cast<const T*>(in_dptr), reinterpret_cast<T*>(out_dptr));
  } else {
    ToContiguousForwardGpu<T, IndexType>
      <<<num_blocks, num_threads, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
          count, block_size, ndim, reinterpret_cast<const T*>(in_dptr), reinterpret_cast<T*>(out_dptr));
  }

}

}  // namespace

template<typename T>
struct ToContiguousUtil<DeviceType::kCUDA, T> : ToContiguousUtilBase {
  using ToContiguousUtilBase::ToContiguousUtilBase;
  static constexpr size_t dsize = sizeof(T);
  void operator()() {
    int constant_memory_size = 0;
    // get device constant memory capacity, for RTX 2080, constant_memory_size is 65536(64kb)
    hipDeviceGetAttribute(&constant_memory_size, hipDeviceAttributeTotalConstantMemory, 0);
    const size_t ndims = contiguous_dim + 1;
    if (ndims == 0) {
      // 0-dim tensor
      OF_CUDA_CHECK(hipMemcpyAsync(out_dptr, in_dptr, block_size * dsize, hipMemcpyDeviceToDevice,
                                    stream->As<ep::CudaStream>()->hip_stream()));
    } else {
      bool is_same = true;
      for (int64_t i = contiguous_dim; i != -1; --i) {
        if (out_stride[i] != in_stride[i]) {
          is_same = false;
          break;
        }
      }
      if (is_same) {
        // if input tensor's strides equals to output's, than just copy one memory-contiguous tensor
        OF_CUDA_CHECK(hipMemcpyAsync(out_dptr, in_dptr, element_count * dsize, hipMemcpyDeviceToDevice,
                                      stream->As<ep::CudaStream>()->hip_stream()));
      } else {
        constexpr size_t pack_size = cuda::elementwise::PackSize<T>();
        if (element_count < GetMaxVal<int32_t>()) {
          LaunchToContiguousKernel<T, int32_t, pack_size>(stream, element_count, ndims, block_size,
                                                          in_stride, out_stride, in_dptr, out_dptr);
        } else {
          LaunchToContiguousKernel<T, int64_t, pack_size>(stream, element_count, ndims, block_size,
                                                          in_stride, out_stride, in_dptr, out_dptr);
        }
      }
    }
  }
};

#define INSTANTIATE_TO_CONTIGUOUS_UTILS_FOR_CUDA(T) \
  template struct ToContiguousUtil<DeviceType::kCUDA, T>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_TO_CONTIGUOUS_UTILS_FOR_CUDA,
                     TO_CONTIGUOUS_TYPES TO_CONTIGUOUS_CUDA_SPECIAL_TYPE)

}  // namespace oneflow
