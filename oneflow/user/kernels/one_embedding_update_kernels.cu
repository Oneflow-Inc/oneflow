#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/user/kernels/model_update_kernel_util.h"
#include "oneflow/core/embedding/embedding_manager.h"

namespace oneflow {

namespace {

template<typename T, typename G, typename IDX>
__global__ void SGDUpdateKernel(const int64_t embedding_size, T scale, float l1, float l2,
                                float weight_decay, float learning_rate_val,
                                const IDX* num_unique_ids, const float* learning_rate,
                                const T* scale_by_ptr, const T* down_scale_by_ptr,
                                const int64_t* skip_if, const G* model_diff, const T* model,
                                T* updated_model) {
  if (skip_if != nullptr && *skip_if != 0) {
    const int64_t n = *num_unique_ids * embedding_size;
    CUDA_1D_KERNEL_LOOP(i, n) { updated_model[i] = model[i]; }
  } else {
    if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
    if (down_scale_by_ptr != nullptr) { scale /= *down_scale_by_ptr; }
    if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
    const int64_t n = *num_unique_ids * embedding_size;
    CUDA_1D_KERNEL_LOOP(i, n) {
      updated_model[i] = model[i];
      SGDUpdateFunctor<T, G>()(model_diff + i, updated_model + i, scale, l1, l2, weight_decay,
                               learning_rate_val);
    }
  }
}

__device__ void GetMomentumOffset(const int32_t line_size, const int32_t embedding_size,
                                  int64_t model_diff_offset, int64_t* model_offset,
                                  int64_t* momentum_offset) {
  const int32_t row = model_diff_offset / embedding_size;
  const int32_t col = model_diff_offset - row * embedding_size;
  *model_offset = row * line_size + col;
  *momentum_offset = *model_offset + embedding_size;
}

template<typename T, typename G, typename IDX>
__global__ void MomentumUpdateKernel(const int64_t line_size, const int64_t embedding_size, T scale,
                                     float l1, float l2, float weight_decay, float beta,
                                     float dampening, bool nesterov, bool maximize,
                                     float learning_rate_val, const IDX* num_unique_ids,
                                     const float* learning_rate, const T* scale_by_ptr,
                                     const T* down_scale_by_ptr, const int64_t* skip_if,
                                     const G* model_diff, const T* unique_values,
                                     T* updated_unique_values) {
  if (skip_if != nullptr && *skip_if != 0) {
    const int64_t n = *num_unique_ids * line_size;
    CUDA_1D_KERNEL_LOOP(i, n) { updated_unique_values[i] = unique_values[i]; }
  } else {
    if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
    if (down_scale_by_ptr != nullptr) { scale /= *down_scale_by_ptr; }
    if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
    const int64_t n = *num_unique_ids * embedding_size;
    CUDA_1D_KERNEL_LOOP(i, n) {
      int64_t model_offset;
      int64_t momentum_offset;
      GetMomentumOffset(line_size, embedding_size, i, &model_offset, &momentum_offset);
      updated_unique_values[model_offset] = unique_values[model_offset];
      updated_unique_values[momentum_offset] = unique_values[momentum_offset];
      MomentumUpdateFunctor<T, G>()(model_diff + i, updated_unique_values + model_offset,
                                    updated_unique_values + momentum_offset, scale, l1, l2, beta,
                                    dampening, nesterov, maximize, weight_decay, learning_rate_val);
    }
  }
}

__device__ void GetAdamOffset(const int32_t line_size, const int32_t embedding_size,
                              int64_t model_diff_offset, int64_t* model_offset, int64_t* m_offset,
                              int64_t* v_offset) {
  const int32_t row = model_diff_offset / embedding_size;
  const int32_t col = model_diff_offset - row * embedding_size;
  *model_offset = row * line_size + col;
  *m_offset = *model_offset + embedding_size;
  *v_offset = *model_offset + 2 * embedding_size;
}

template<typename T, typename G, typename IDX>
__global__ void AdamUpdateKernel(const int32_t line_size, const int32_t embedding_size, T scale,
                                 float l1, float l2, float weight_decay, float beta1, float beta2,
                                 float epsilon, float learning_rate_val, float bias_correction1_val,
                                 float bias_correction2_val, const float* bias_correction1_ptr,
                                 const float* bias_correction2_ptr, const IDX* num_unique_ids,
                                 const float* learning_rate, const T* scale_by_ptr,
                                 const T* down_scale_by_ptr, const int64_t* skip_if,
                                 const G* model_diff, const T* unique_values,
                                 T* updated_unique_values) {
  if (skip_if != nullptr && *skip_if != 0) {
    const int64_t n = *num_unique_ids * line_size;
    CUDA_1D_KERNEL_LOOP(i, n) {
      // The n is the unique_values elem_cnt, so not need to use GetAdamOffset.
      updated_unique_values[i] = unique_values[i];
    }
  } else {
    if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
    if (down_scale_by_ptr != nullptr) { scale /= *down_scale_by_ptr; }
    if (bias_correction1_ptr != nullptr) { bias_correction1_val = *bias_correction1_ptr; }
    if (bias_correction2_ptr != nullptr) { bias_correction2_val = *bias_correction2_ptr; }
    if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
    const int64_t n = *num_unique_ids * embedding_size;
    // The n is model_diff elem_cnt.
    CUDA_1D_KERNEL_LOOP(i, n) {
      int64_t model_offset;
      int64_t m_offset;
      int64_t v_offset;
      GetAdamOffset(line_size, embedding_size, i, &model_offset, &m_offset, &v_offset);
      updated_unique_values[model_offset] = unique_values[model_offset];
      updated_unique_values[m_offset] = unique_values[m_offset];
      updated_unique_values[v_offset] = unique_values[v_offset];
      AdamUpdateFunctor<T, G>()(model_diff + i, updated_unique_values + model_offset,
                                updated_unique_values + m_offset, updated_unique_values + v_offset,
                                nullptr, scale, l1, l2, beta1, beta2, epsilon, weight_decay, false,
                                bias_correction1_val, bias_correction2_val, learning_rate_val);
    }
  }
}

// Note(guoran): The SmartDecaySparseAdam is from
// https://github.com/pytorch/pytorch/blob/master/caffe2/sgd/adam_op.h#L57
template<typename T, typename G, typename IDX>
__global__ void SmartDecaySparseAdamUpdateKernel(
    const int32_t line_size, const int32_t embedding_size, T scale, float l1, float l2,
    float weight_decay, float beta1, float beta2, float epsilon, float learning_rate_val,
    int64_t step_col_offset, const IDX* num_unique_ids, const float* learning_rate,
    const int64_t* train_step_ptr, const T* scale_by_ptr, const T* down_scale_by_ptr,
    const int64_t* skip_if, const G* model_diff, const T* unique_values, T* updated_unique_values) {
  if (skip_if != nullptr && *skip_if != 0) {
    const int64_t n = *num_unique_ids * line_size;
    CUDA_1D_KERNEL_LOOP(i, n) {
      // The n is the unique_values elem_cnt, so not need to use GetAdamOffset.
      updated_unique_values[i] = unique_values[i];
    }
  } else {
    if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
    if (down_scale_by_ptr != nullptr) { scale /= *down_scale_by_ptr; }
    if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
    const int64_t n = *num_unique_ids * embedding_size;
    // The n is model_diff elem_cnt.
    CUDA_1D_KERNEL_LOOP(i, n) {
      const int32_t row = i / embedding_size;
      const int32_t col = i - row * embedding_size;
      int64_t model_offset = row * line_size + col;
      int64_t m_offset = model_offset + embedding_size;
      int64_t v_offset = model_offset + 2 * embedding_size;
      int64_t step_offset = row * line_size + step_col_offset;
      const T model_val = *(unique_values + model_offset);
      const T m_val = *(unique_values + m_offset);
      const T v_val = *(unique_values + v_offset);
      T model_diff_t =
          CastScaleRegularizeGradientFunctor<T, G>()(*(model_diff + i), model_val, scale, l1, l2);
      int64_t prev_step = *reinterpret_cast<const int64_t*>(unique_values + step_offset);
      int64_t cur_step = *train_step_ptr + 1;
      int64_t skip_step = cur_step - prev_step;
      float catchup = 0.0;
      if (skip_step > 1) {
        catchup = m_val * beta1 * (1 - pow(beta1, skip_step - 1)) / (1 - beta1);
      }
      const T next_m = pow(beta1, skip_step) * m_val + (1 - beta1) * model_diff_t;
      const T next_v = pow(beta2, skip_step) * v_val + (1 - beta2) * model_diff_t * model_diff_t;
      updated_unique_values[m_offset] = next_m;
      updated_unique_values[v_offset] = next_v;
      updated_unique_values[model_offset] =
          model_val - (learning_rate_val * (next_m + catchup)) / (sqrt(next_v) + epsilon);
      if (col == 0) { *reinterpret_cast<int64_t*>(updated_unique_values + step_offset) = cur_step; }
    }
  }
}

template<typename T, typename G, typename IDX>
__global__ void AdagradUpdateKernel(const int64_t line_size, const int64_t embedding_size, T scale,
                                    float l1, float l2, float weight_decay, float lr_decay,
                                    float epsilon, float learning_rate_val, int64_t train_step,
                                    const IDX* num_unique_ids, const float* learning_rate,
                                    const int64_t* train_step_ptr, const T* scale_by_ptr,
                                    const T* down_scale_by_ptr, const int64_t* skip_if,
                                    const G* model_diff, const T* unique_values,
                                    T* updated_unique_values) {
  if (skip_if != nullptr && *skip_if != 0) {
    const int64_t n = *num_unique_ids * line_size;
    CUDA_1D_KERNEL_LOOP(i, n) { updated_unique_values[i] = unique_values[i]; }
  } else {
    if (train_step_ptr != nullptr) { train_step = *train_step_ptr + 1; }
    if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
    if (down_scale_by_ptr != nullptr) { scale /= *down_scale_by_ptr; }
    if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
    learning_rate_val = learning_rate_val / (1 + (train_step - 1) * lr_decay);
    const int64_t n = *num_unique_ids * embedding_size;
    CUDA_1D_KERNEL_LOOP(i, n) {
      int64_t model_offset;
      int64_t sum_offset;
      GetMomentumOffset(line_size, embedding_size, i, &model_offset, &sum_offset);
      updated_unique_values[model_offset] = unique_values[model_offset];
      updated_unique_values[sum_offset] = unique_values[sum_offset];
      AdagradUpdateFunctor<T, G>()(model_diff + i, updated_unique_values + model_offset,
                                   updated_unique_values + sum_offset, scale, l1, l2, epsilon,
                                   weight_decay, learning_rate_val);
    }
  }
}

__device__ void GetFtrlOffset(const int32_t line_size, const int32_t embedding_size,
                              int64_t model_diff_offset, int64_t* model_offset,
                              int64_t* accumulate_offset, int64_t* z_offset) {
  const int32_t row = model_diff_offset / embedding_size;
  const int32_t col = model_diff_offset - row * embedding_size;
  *model_offset = row * line_size + col;
  *accumulate_offset = *model_offset + embedding_size;
  *z_offset = *model_offset + 2 * embedding_size;
}

template<typename T, typename G, typename IDX>
__global__ void FtrlUpdateKernel(const int32_t line_size, const int32_t embedding_size, T scale,
                                 float l1, float l2, float weight_decay, float lr_power,
                                 float lambda1, float lambda2, float beta, float learning_rate_val,
                                 const IDX* num_unique_ids, const float* learning_rate,
                                 const T* scale_by_ptr, const T* down_scale_by_ptr,
                                 const int64_t* skip_if, const G* model_diff,
                                 const T* unique_values, T* updated_unique_values) {
  if (skip_if != nullptr && *skip_if != 0) {
    const int64_t n = *num_unique_ids * line_size;
    CUDA_1D_KERNEL_LOOP(i, n) { updated_unique_values[i] = unique_values[i]; }
  } else {
    if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
    if (down_scale_by_ptr != nullptr) { scale /= *down_scale_by_ptr; }
    if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
    const int64_t n = *num_unique_ids * embedding_size;
    CUDA_1D_KERNEL_LOOP(i, n) {
      int64_t model_offset;
      int64_t accumulate_offset;
      int64_t z_offset;
      GetFtrlOffset(line_size, embedding_size, i, &model_offset, &accumulate_offset, &z_offset);
      updated_unique_values[model_offset] = unique_values[model_offset];
      updated_unique_values[accumulate_offset] = unique_values[accumulate_offset];
      updated_unique_values[z_offset] = unique_values[z_offset];
      FtrlUpdateFunctor<T, G>()(model_diff + i, updated_unique_values + model_offset,
                                updated_unique_values + accumulate_offset,
                                updated_unique_values + z_offset, scale, l1, l2, lr_power, lambda1,
                                lambda2, beta, weight_decay, learning_rate_val);
    }
  }
}

class EmbeddingUpdateKernelState final : public user_op::OpKernelState {
 public:
  explicit EmbeddingUpdateKernelState(user_op::KernelInitContext* ctx) {
    const std::string& embedding_name = ctx->Attr<std::string>("embedding_name");
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    embedding_state_ = Singleton<embedding::EmbeddingManager>::Get()->GetEmbeddingState(
        embedding_name, parallel_id);
  }
  ~EmbeddingUpdateKernelState() override = default;

  embedding::EmbeddingState* EmbeddingState() { return embedding_state_; }

 private:
  embedding::EmbeddingState* embedding_state_;
};

}  // namespace

template<typename T, typename G, typename IDX>
class SgdEmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  SgdEmbeddingUpdateKernel() = default;
  ~SgdEmbeddingUpdateKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingUpdateKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingUpdateKernelState*>(state);
    CHECK(kernel_state != nullptr);
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingUpdateStart(ctx, current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    CHECK_EQ(embedding_grad->shape_view().NumAxes(), 2);
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    CHECK_EQ(line_size, embedding_size);
    const auto scale = ctx->Attr<double>("scale");
    const float l1 = ctx->Attr<float>("l1");
    const float l2 = ctx->Attr<float>("l2");
    const auto weight_decay = ctx->Attr<float>("weight_decay");
    const float learning_rate_val = ctx->Attr<float>("learning_rate_val");
    const float* learning_rate_ptr = nullptr;
    if (ctx->has_input("learning_rate", 0)) {
      const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
      learning_rate_ptr = learning_rate->dptr<float>();
    }
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->shape_view().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const T* down_scale_by_ptr = nullptr;
    if (ctx->has_input("down_scale_by_tensor", 0)) {
      const user_op::Tensor* down_scale_by_tensor =
          ctx->Tensor4ArgNameAndIndex("down_scale_by_tensor", 0);
      CHECK_EQ(down_scale_by_tensor->shape_view().elem_cnt(), 1);
      down_scale_by_ptr = down_scale_by_tensor->dptr<T>();
    }
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape_view().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }
    // update kernel
    const T* unique_embeddings_ptr =
        reinterpret_cast<const T*>(embedding_state->EmbeddingUpdateUniqueEmbeddings(current_iter_));
    T* updated_unique_embeddings_ptr = reinterpret_cast<T*>(
        embedding_state->EmbeddingUpdateUpdatedUniqueEmbeddings(current_iter_));
    const uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    const int64_t embedding_grad_elem_cnt = num_unique * embedding_size;
    SGDUpdateKernel<T, G, IDX>
        <<<BlocksNum4ThreadsNum(embedding_grad_elem_cnt), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            embedding_size, scale, l1, l2, weight_decay, learning_rate_val,
            reinterpret_cast<const IDX*>(num_unique_ids->dptr()), learning_rate_ptr, scale_by_ptr,
            down_scale_by_ptr, skip_if_ptr, embedding_grad->dptr<G>(), unique_embeddings_ptr,
            updated_unique_embeddings_ptr);
    embedding_state->OnEmbeddingUpdateEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define IDX_DATA_TYPE_SEQ                           \
  OF_PP_MAKE_TUPLE_SEQ(uint32_t, DataType::kUInt32) \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32)

#define REGISTER_CUDA_ONE_EMBEDDING_SGD_UPDATE_KERNEL(t_dtype_pair, g_type_pair, idx_dtype_pair)  \
  REGISTER_USER_KERNEL("one_embedding_sgd_update")                                                \
      .SetCreateFn<                                                                               \
          SgdEmbeddingUpdateKernel<OF_PP_PAIR_FIRST(t_dtype_pair), OF_PP_PAIR_FIRST(g_type_pair), \
                                   OF_PP_PAIR_FIRST(idx_dtype_pair)>>()                           \
      .SetIsMatchedHob(                                                                           \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                         \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))     \
          && (user_op::HobDataType("embedding_grad", 0) == OF_PP_PAIR_SECOND(g_type_pair))        \
          && (user_op::HobDataType("unique_embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair)));

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_ONE_EMBEDDING_SGD_UPDATE_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ,
                                 IDX_DATA_TYPE_SEQ)

template<typename T, typename G, typename IDX>
class MomentumEmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  MomentumEmbeddingUpdateKernel() : current_iter_(0){};
  ~MomentumEmbeddingUpdateKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingUpdateKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingUpdateKernelState*>(state);
    CHECK(kernel_state != nullptr);
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingUpdateStart(ctx, current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    CHECK_EQ(embedding_grad->shape_view().NumAxes(), 2);
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    CHECK_EQ(line_size, embedding_size * 2);
    const float l1 = ctx->Attr<float>("l1");
    const float l2 = ctx->Attr<float>("l2");
    const auto weight_decay = ctx->Attr<float>("weight_decay");
    const auto beta = ctx->Attr<float>("beta");
    // TODO: Suppoprt dampening, nesterov, maximize in OneEmbeddingMomentumUpdate(zhengzekang).
    const float dampening = 0.0;
    const bool nesterov = false;
    const bool maximize = false;
    const auto scale = ctx->Attr<double>("scale");
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->shape_view().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const T* down_scale_by_ptr = nullptr;
    if (ctx->has_input("down_scale_by_tensor", 0)) {
      const user_op::Tensor* down_scale_by_tensor =
          ctx->Tensor4ArgNameAndIndex("down_scale_by_tensor", 0);
      CHECK_EQ(down_scale_by_tensor->shape_view().elem_cnt(), 1);
      down_scale_by_ptr = down_scale_by_tensor->dptr<T>();
    }
    const float learning_rate_val = ctx->Attr<float>("learning_rate_val");
    const float* learning_rate_ptr = nullptr;
    if (ctx->has_input("learning_rate", 0)) {
      const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
      learning_rate_ptr = learning_rate->dptr<float>();
    }
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape_view().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }
    // update kernel
    const T* unique_embeddings_ptr =
        reinterpret_cast<const T*>(embedding_state->EmbeddingUpdateUniqueEmbeddings(current_iter_));
    T* updated_unique_embeddings_ptr = reinterpret_cast<T*>(
        embedding_state->EmbeddingUpdateUpdatedUniqueEmbeddings(current_iter_));
    const uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    const int64_t embedding_grad_elem_cnt = num_unique * embedding_size;
    MomentumUpdateKernel<T, G, IDX>
        <<<BlocksNum4ThreadsNum(embedding_grad_elem_cnt), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            line_size, embedding_size, scale, l1, l2, weight_decay, beta, dampening, nesterov,
            maximize, learning_rate_val, reinterpret_cast<const IDX*>(num_unique_ids->dptr()),
            learning_rate_ptr, scale_by_ptr, down_scale_by_ptr, skip_if_ptr,
            embedding_grad->dptr<G>(), unique_embeddings_ptr, updated_unique_embeddings_ptr);
    embedding_state->OnEmbeddingUpdateEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define REGISTER_CUDA_ONE_EMBEDDING_MOMENTUM_UPDATE_KERNEL(t_dtype_pair, g_type_pair,         \
                                                           idx_dtype_pair)                    \
  REGISTER_USER_KERNEL("one_embedding_momentum_update")                                       \
      .SetCreateFn<MomentumEmbeddingUpdateKernel<OF_PP_PAIR_FIRST(t_dtype_pair),              \
                                                 OF_PP_PAIR_FIRST(g_type_pair),               \
                                                 OF_PP_PAIR_FIRST(idx_dtype_pair)>>()         \
      .SetIsMatchedHob(                                                                       \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                     \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair)) \
          && (user_op::HobDataType("embedding_grad", 0) == OF_PP_PAIR_SECOND(g_type_pair))    \
          && (user_op::HobDataType("unique_embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair)));

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_ONE_EMBEDDING_MOMENTUM_UPDATE_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ,
                                 IDX_DATA_TYPE_SEQ)

template<typename T, typename G, typename IDX>
class AdamEmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  AdamEmbeddingUpdateKernel() : current_iter_(0){};
  ~AdamEmbeddingUpdateKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingUpdateKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingUpdateKernelState*>(state);
    CHECK(kernel_state != nullptr);
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingUpdateStart(ctx, current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_embeddings = ctx->Tensor4ArgNameAndIndex("unique_embeddings", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    user_op::Tensor* updated_unique_embeddings =
        ctx->Tensor4ArgNameAndIndex("updated_unique_embeddings", 0);
    CHECK_EQ(embedding_grad->shape_view().NumAxes(), 2);
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    CHECK_EQ(line_size, embedding_size * 3);

    const float l1 = ctx->Attr<float>("l1");
    const float l2 = ctx->Attr<float>("l2");
    const auto weight_decay = ctx->Attr<float>("weight_decay");
    const auto beta1 = ctx->Attr<float>("beta1");
    const auto beta2 = ctx->Attr<float>("beta2");
    const auto epsilon = ctx->Attr<float>("epsilon");
    const bool do_bias_correction = ctx->Attr<bool>("do_bias_correction");
    const auto scale = ctx->Attr<double>("scale");
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->shape_view().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const T* down_scale_by_ptr = nullptr;
    if (ctx->has_input("down_scale_by_tensor", 0)) {
      const user_op::Tensor* down_scale_by_tensor =
          ctx->Tensor4ArgNameAndIndex("down_scale_by_tensor", 0);
      CHECK_EQ(down_scale_by_tensor->shape_view().elem_cnt(), 1);
      down_scale_by_ptr = down_scale_by_tensor->dptr<T>();
    }
    const float learning_rate_val = ctx->Attr<float>("learning_rate_val");
    const float* learning_rate_ptr = nullptr;
    if (ctx->has_input("learning_rate", 0)) {
      const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
      learning_rate_ptr = learning_rate->dptr<float>();
    }
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape_view().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }
    const float bias_correction1_val = ctx->Attr<float>("bias_correction1_val");
    const float* bias_correction1_ptr = nullptr;
    if (ctx->has_input("bias_correction1", 0)) {
      bias_correction1_ptr = ctx->Tensor4ArgNameAndIndex("bias_correction1", 0)->dptr<float>();
    }
    const float bias_correction2_val = ctx->Attr<float>("bias_correction2_val");
    const float* bias_correction2_ptr = nullptr;
    if (ctx->has_input("bias_correction2", 0)) {
      bias_correction2_ptr = ctx->Tensor4ArgNameAndIndex("bias_correction2", 0)->dptr<float>();
    }
    // update kernel
    const T* unique_embeddings_ptr =
        reinterpret_cast<const T*>(embedding_state->EmbeddingUpdateUniqueEmbeddings(current_iter_));
    T* updated_unique_embeddings_ptr = reinterpret_cast<T*>(
        embedding_state->EmbeddingUpdateUpdatedUniqueEmbeddings(current_iter_));
    const uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    const int64_t embedding_grad_elem_cnt = num_unique * embedding_size;
    AdamUpdateKernel<T, G, IDX>
        <<<BlocksNum4ThreadsNum(embedding_grad_elem_cnt), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            line_size, embedding_size, static_cast<T>(scale), l1, l2, weight_decay, beta1, beta2,
            epsilon, learning_rate_val, bias_correction1_val, bias_correction2_val,
            bias_correction1_ptr, bias_correction2_ptr,
            reinterpret_cast<const IDX*>(num_unique_ids->dptr()), learning_rate_ptr, scale_by_ptr,
            down_scale_by_ptr, skip_if_ptr, embedding_grad->dptr<G>(), unique_embeddings_ptr,
            updated_unique_embeddings_ptr);
    embedding_state->OnEmbeddingUpdateEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define REGISTER_CUDA_ONE_EMBEDDING_ADAM_UPDATE_KERNEL(t_dtype_pair, g_type_pair, idx_dtype_pair)  \
  REGISTER_USER_KERNEL("one_embedding_adam_update")                                                \
      .SetCreateFn<                                                                                \
          AdamEmbeddingUpdateKernel<OF_PP_PAIR_FIRST(t_dtype_pair), OF_PP_PAIR_FIRST(g_type_pair), \
                                    OF_PP_PAIR_FIRST(idx_dtype_pair)>>()                           \
      .SetIsMatchedHob(                                                                            \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                          \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))      \
          && (user_op::HobDataType("embedding_grad", 0) == OF_PP_PAIR_SECOND(g_type_pair))         \
          && (user_op::HobDataType("unique_embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair)));

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_ONE_EMBEDDING_ADAM_UPDATE_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ,
                                 IDX_DATA_TYPE_SEQ)

template<typename T, typename G, typename IDX>
class SmartDecaySparseAdamEmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  SmartDecaySparseAdamEmbeddingUpdateKernel() : current_iter_(0){};
  ~SmartDecaySparseAdamEmbeddingUpdateKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingUpdateKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingUpdateKernelState*>(state);
    CHECK(kernel_state != nullptr);
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingUpdateStart(ctx, current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    user_op::Tensor* updated_unique_embeddings =
        ctx->Tensor4ArgNameAndIndex("updated_unique_embeddings", 0);
    CHECK_EQ(embedding_grad->shape_view().NumAxes(), 2);
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    const float l1 = ctx->Attr<float>("l1");
    const float l2 = ctx->Attr<float>("l2");
    const auto weight_decay = ctx->Attr<float>("weight_decay");
    const auto beta1 = ctx->Attr<float>("beta1");
    const auto beta2 = ctx->Attr<float>("beta2");
    const auto epsilon = ctx->Attr<float>("epsilon");
    const bool do_bias_correction = ctx->Attr<bool>("do_bias_correction");
    const auto scale = ctx->Attr<double>("scale");
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->shape_view().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const T* down_scale_by_ptr = nullptr;
    if (ctx->has_input("down_scale_by_tensor", 0)) {
      const user_op::Tensor* down_scale_by_tensor =
          ctx->Tensor4ArgNameAndIndex("down_scale_by_tensor", 0);
      CHECK_EQ(down_scale_by_tensor->shape_view().elem_cnt(), 1);
      down_scale_by_ptr = down_scale_by_tensor->dptr<T>();
    }
    const float learning_rate_val = ctx->Attr<float>("learning_rate_val");
    const float* learning_rate_ptr = nullptr;
    if (ctx->has_input("learning_rate", 0)) {
      const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
      learning_rate_ptr = learning_rate->dptr<float>();
    }
    const int64_t train_step_val = ctx->Attr<int64_t>("train_step_val");
    const int64_t* train_step_ptr = nullptr;
    if (ctx->has_input("train_step", 0)) {
      const user_op::Tensor* train_step = ctx->Tensor4ArgNameAndIndex("train_step", 0);
      train_step_ptr = train_step->dptr<int64_t>();
    }
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape_view().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }
    // update kernel
    const T* unique_embeddings_ptr =
        reinterpret_cast<const T*>(embedding_state->EmbeddingUpdateUniqueEmbeddings(current_iter_));
    T* updated_unique_embeddings_ptr = reinterpret_cast<T*>(
        embedding_state->EmbeddingUpdateUpdatedUniqueEmbeddings(current_iter_));
    const uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    const int64_t embedding_grad_elem_cnt = num_unique * embedding_size;

    const int64_t value_dtype_size = GetSizeOfDataType(updated_unique_embeddings->data_type());
    const int64_t step_dtype_size = sizeof(int64_t);
    const int64_t model_and_states_bytes = embedding_size * 3 * value_dtype_size;
    const int64_t align_to_step_size_bytes =
        (model_and_states_bytes + step_dtype_size - 1) / step_dtype_size * step_dtype_size;
    const int64_t step_col_offset = align_to_step_size_bytes / value_dtype_size;
    const int64_t smart_decay_sparse_adam_line_size =
        (align_to_step_size_bytes + step_dtype_size) / value_dtype_size;
    CHECK_EQ(line_size, smart_decay_sparse_adam_line_size);

    SmartDecaySparseAdamUpdateKernel<T, G, IDX>
        <<<BlocksNum4ThreadsNum(embedding_grad_elem_cnt), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            line_size, embedding_size, static_cast<T>(scale), l1, l2, weight_decay, beta1, beta2,
            epsilon, learning_rate_val, step_col_offset,
            reinterpret_cast<const IDX*>(num_unique_ids->dptr()), learning_rate_ptr, train_step_ptr,
            scale_by_ptr, down_scale_by_ptr, skip_if_ptr, embedding_grad->dptr<G>(),
            unique_embeddings_ptr, updated_unique_embeddings_ptr);
    embedding_state->OnEmbeddingUpdateEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define REGISTER_CUDA_ONE_EMBEDDING_SMART_DECAY_SPARSE_ADAM_UPDATE_KERNEL(                        \
    t_dtype_pair, g_type_pair, idx_dtype_pair)                                                    \
  REGISTER_USER_KERNEL("one_embedding_smart_decay_sparse_adam_update")                            \
      .SetCreateFn<SmartDecaySparseAdamEmbeddingUpdateKernel<OF_PP_PAIR_FIRST(t_dtype_pair),      \
                                                             OF_PP_PAIR_FIRST(g_type_pair),       \
                                                             OF_PP_PAIR_FIRST(idx_dtype_pair)>>() \
      .SetIsMatchedHob(                                                                           \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                         \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))     \
          && (user_op::HobDataType("embedding_grad", 0) == OF_PP_PAIR_SECOND(g_type_pair))        \
          && (user_op::HobDataType("unique_embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair)));

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_ONE_EMBEDDING_SMART_DECAY_SPARSE_ADAM_UPDATE_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ,
                                 IDX_DATA_TYPE_SEQ)

template<typename T, typename G, typename IDX>
class AdagradEmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  AdagradEmbeddingUpdateKernel() : current_iter_(0){};
  ~AdagradEmbeddingUpdateKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingUpdateKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingUpdateKernelState*>(state);
    CHECK(kernel_state != nullptr);
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingUpdateStart(ctx, current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_embeddings = ctx->Tensor4ArgNameAndIndex("unique_embeddings", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    user_op::Tensor* updated_unique_embeddings =
        ctx->Tensor4ArgNameAndIndex("updated_unique_embeddings", 0);
    CHECK_EQ(embedding_grad->shape_view().NumAxes(), 2);
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    CHECK_EQ(line_size, embedding_size * 2);
    const float l1 = ctx->Attr<float>("l1");
    const float l2 = ctx->Attr<float>("l2");
    const auto weight_decay = ctx->Attr<float>("weight_decay");
    const auto lr_decay = ctx->Attr<float>("lr_decay");
    const auto epsilon = ctx->Attr<float>("epsilon");
    const auto scale = ctx->Attr<double>("scale");
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->shape_view().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const T* down_scale_by_ptr = nullptr;
    if (ctx->has_input("down_scale_by_tensor", 0)) {
      const user_op::Tensor* down_scale_by_tensor =
          ctx->Tensor4ArgNameAndIndex("down_scale_by_tensor", 0);
      CHECK_EQ(down_scale_by_tensor->shape_view().elem_cnt(), 1);
      down_scale_by_ptr = down_scale_by_tensor->dptr<T>();
    }
    const float learning_rate_val = ctx->Attr<float>("learning_rate_val");
    const float* learning_rate_ptr = nullptr;
    if (ctx->has_input("learning_rate", 0)) {
      const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
      learning_rate_ptr = learning_rate->dptr<float>();
    }
    const int64_t train_step_val = ctx->Attr<int64_t>("train_step_val");
    const int64_t* train_step_ptr = nullptr;
    if (ctx->has_input("train_step", 0)) {
      const user_op::Tensor* train_step = ctx->Tensor4ArgNameAndIndex("train_step", 0);
      train_step_ptr = train_step->dptr<int64_t>();
    }
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape_view().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }
    // update kernel
    const T* unique_embeddings_ptr =
        reinterpret_cast<const T*>(embedding_state->EmbeddingUpdateUniqueEmbeddings(current_iter_));
    T* updated_unique_embeddings_ptr = reinterpret_cast<T*>(
        embedding_state->EmbeddingUpdateUpdatedUniqueEmbeddings(current_iter_));
    const uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    const int64_t embedding_grad_elem_cnt = num_unique * embedding_size;
    AdagradUpdateKernel<T, G, IDX>
        <<<BlocksNum4ThreadsNum(embedding_grad_elem_cnt), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            line_size, embedding_size, static_cast<T>(scale), l1, l2, weight_decay, lr_decay,
            epsilon, learning_rate_val, train_step_val,
            reinterpret_cast<const IDX*>(num_unique_ids->dptr()), learning_rate_ptr, train_step_ptr,
            scale_by_ptr, down_scale_by_ptr, skip_if_ptr, embedding_grad->dptr<G>(),
            unique_embeddings_ptr, updated_unique_embeddings_ptr);
    embedding_state->OnEmbeddingUpdateEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};

#define REGISTER_CUDA_ONE_EMBEDDING_ADAGRAD_UPDATE_KERNEL(t_dtype_pair, g_type_pair,          \
                                                          idx_dtype_pair)                     \
  REGISTER_USER_KERNEL("one_embedding_adagrad_update")                                        \
      .SetCreateFn<AdagradEmbeddingUpdateKernel<OF_PP_PAIR_FIRST(t_dtype_pair),               \
                                                OF_PP_PAIR_FIRST(g_type_pair),                \
                                                OF_PP_PAIR_FIRST(idx_dtype_pair)>>()          \
      .SetIsMatchedHob(                                                                       \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                     \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair)) \
          && (user_op::HobDataType("embedding_grad", 0) == OF_PP_PAIR_SECOND(g_type_pair))    \
          && (user_op::HobDataType("unique_embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair)));

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_ONE_EMBEDDING_ADAGRAD_UPDATE_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ,
                                 IDX_DATA_TYPE_SEQ)

template<typename T, typename G, typename IDX>
class FtrlEmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  FtrlEmbeddingUpdateKernel() : current_iter_(0){};
  ~FtrlEmbeddingUpdateKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingUpdateKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingUpdateKernelState*>(state);
    CHECK(kernel_state != nullptr);
    embedding::EmbeddingState* embedding_state = kernel_state->EmbeddingState();
    embedding_state->OnEmbeddingUpdateStart(ctx, current_iter_);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    CHECK_EQ(embedding_grad->shape_view().NumAxes(), 2)
        << "The NumAxes of embedding_grad should be equal to 2. ";
    const int64_t line_size = ctx->Attr<int64_t>("line_size");
    const int64_t embedding_size = ctx->Attr<int64_t>("embedding_size");
    CHECK_EQ(line_size, embedding_size * 3)
        << "The line_size should be equal to 3 x embedding_size. ";
    const float l1 = 0.0;
    const float l2 = 0.0;
    const float weight_decay = ctx->Attr<float>("weight_decay");
    // TODO(zhengzekang): Undefined behavior for ftrl optimizer with weight_decay in `abs(new_z_val)
    // < lambda1` condition.
    CHECK_EQ(weight_decay, static_cast<float>(0.0))
        << "Currently not support for setting weight decay. ";
    const float lr_power = ctx->Attr<float>("lr_power");
    const float lambda1 = ctx->Attr<float>("lambda1");
    const float lambda2 = ctx->Attr<float>("lambda2");
    const float beta = ctx->Attr<float>("beta");
    const double scale = ctx->Attr<double>("scale");
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->shape_view().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const T* down_scale_by_ptr = nullptr;
    if (ctx->has_input("down_scale_by_tensor", 0)) {
      const user_op::Tensor* down_scale_by_tensor =
          ctx->Tensor4ArgNameAndIndex("down_scale_by_tensor", 0);
      CHECK_EQ(down_scale_by_tensor->shape_view().elem_cnt(), 1);
      down_scale_by_ptr = down_scale_by_tensor->dptr<T>();
    }
    const float learning_rate_val = ctx->Attr<float>("learning_rate_val");
    const float* learning_rate_ptr = nullptr;
    if (ctx->has_input("learning_rate", 0)) {
      const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
      learning_rate_ptr = learning_rate->dptr<float>();
    }
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape_view().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }
    // update kernel
    const T* unique_embeddings_ptr =
        reinterpret_cast<const T*>(embedding_state->EmbeddingUpdateUniqueEmbeddings(current_iter_));
    T* updated_unique_embeddings_ptr = reinterpret_cast<T*>(
        embedding_state->EmbeddingUpdateUpdatedUniqueEmbeddings(current_iter_));
    const uint32_t num_unique = embedding_state->GetIdNumUnique(current_iter_);
    const int64_t embedding_grad_elem_cnt = num_unique * embedding_size;
    FtrlUpdateKernel<T, G, IDX>
        <<<BlocksNum4ThreadsNum(embedding_grad_elem_cnt), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            line_size, embedding_size, static_cast<T>(scale), l1, l2, weight_decay, lr_power,
            lambda1, lambda2, beta, learning_rate_val,
            reinterpret_cast<const IDX*>(num_unique_ids->dptr()), learning_rate_ptr, scale_by_ptr,
            down_scale_by_ptr, skip_if_ptr, embedding_grad->dptr<G>(), unique_embeddings_ptr,
            updated_unique_embeddings_ptr);
    embedding_state->OnEmbeddingUpdateEnd(ctx, current_iter_);
    current_iter_++;
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  mutable int64_t current_iter_;
};
#define REGISTER_CUDA_ONE_EMBEDDING_FTRL_UPDATE_KERNEL(t_dtype_pair, g_type_pair, idx_dtype_pair)  \
  REGISTER_USER_KERNEL("one_embedding_ftrl_update")                                                \
      .SetCreateFn<                                                                                \
          FtrlEmbeddingUpdateKernel<OF_PP_PAIR_FIRST(t_dtype_pair), OF_PP_PAIR_FIRST(g_type_pair), \
                                    OF_PP_PAIR_FIRST(idx_dtype_pair)>>()                           \
      .SetIsMatchedHob(                                                                            \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                          \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))      \
          && (user_op::HobDataType("embedding_grad", 0) == OF_PP_PAIR_SECOND(g_type_pair))         \
          && (user_op::HobDataType("unique_embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair)));
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_ONE_EMBEDDING_FTRL_UPDATE_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ,
                                 IDX_DATA_TYPE_SEQ)

}  // namespace oneflow
