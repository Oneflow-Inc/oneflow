#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/user/kernels/model_update_kernel_util.h"

namespace oneflow {

namespace {

template<typename T, typename G, typename IDX>
__global__ void SGDUpdateKernel(const int64_t embedding_size, T scale, float l1, float l2,
                                float weight_decay, const IDX* num_unique_ids,
                                const float* learning_rate, const T* scale_by_ptr,
                                const T* down_scale_by_ptr, const int64_t* skip_if,
                                const G* model_diff, const T* model, T* updated_model) {
  if (skip_if != nullptr && *skip_if != 0) {
    const int64_t n = *num_unique_ids * embedding_size;
    CUDA_1D_KERNEL_LOOP(i, n) { updated_model[i] = model[i]; }
  } else {
    if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
    if (down_scale_by_ptr != nullptr) { scale /= *down_scale_by_ptr; }
    float learning_rate_val = *learning_rate;
    const int64_t n = *num_unique_ids * embedding_size;
    CUDA_1D_KERNEL_LOOP(i, n) {
      updated_model[i] = model[i];
      SGDUpdateFunctor<T, G>()(model_diff + i, updated_model + i, scale, l1, l2, weight_decay,
                               learning_rate_val);
    }
  }
}

__device__ void GetMomentumOffset(const int32_t line_size, const int32_t embedding_size,
                                  int64_t model_diff_offset, int64_t* model_offset,
                                  int64_t* momentum_offset) {
  const int32_t row = model_diff_offset / embedding_size;
  const int32_t col = model_diff_offset - row * embedding_size;
  *model_offset = row * line_size + col;
  *momentum_offset = *model_offset + embedding_size;
}

template<typename T, typename G, typename IDX>
__global__ void MomentumUpdateKernel(const int64_t line_size, const int64_t embedding_size, T scale,
                                     float l1, float l2, float weight_decay, float beta,
                                     const IDX* num_unique_ids, const float* learning_rate,
                                     const T* scale_by_ptr, const T* down_scale_by_ptr,
                                     const int64_t* skip_if, const G* model_diff,
                                     const T* unique_values, T* updated_unique_values) {
  if (skip_if != nullptr && *skip_if != 0) {
    const int64_t n = *num_unique_ids * line_size;
    CUDA_1D_KERNEL_LOOP(i, n) { updated_unique_values[i] = unique_values[i]; }
  } else {
    if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
    if (down_scale_by_ptr != nullptr) { scale /= *down_scale_by_ptr; }
    float learning_rate_val = *learning_rate;
    const int64_t n = *num_unique_ids * embedding_size;
    CUDA_1D_KERNEL_LOOP(i, n) {
      int64_t model_offset;
      int64_t momentum_offset;
      GetMomentumOffset(line_size, embedding_size, i, &model_offset, &momentum_offset);
      updated_unique_values[model_offset] = unique_values[model_offset];
      updated_unique_values[momentum_offset] = unique_values[momentum_offset];
      MomentumUpdateFunctor<T, G>()(model_diff + i, updated_unique_values + model_offset,
                                    updated_unique_values + momentum_offset, scale, l1, l2, beta,
                                    weight_decay, learning_rate_val);
    }
  }
}

__device__ void GetAdamOffset(const int32_t line_size, const int32_t embedding_size,
                              int64_t model_diff_offset, int64_t* model_offset, int64_t* m_offset,
                              int64_t* v_offset) {
  const int32_t row = model_diff_offset / embedding_size;
  const int32_t col = model_diff_offset - row * embedding_size;
  *model_offset = row * line_size + col;
  *m_offset = *model_offset + embedding_size;
  *v_offset = *model_offset + 2 * embedding_size;
}

template<typename T, typename G, typename IDX>
__global__ void AdamUpdateKernel(const int32_t line_size, const int32_t embedding_size, T scale,
                                 float l1, float l2, float weight_decay, float beta1, float beta2,
                                 float epsilon, const float* bias_correction1_ptr,
                                 const float* bias_correction2_ptr, const IDX* num_unique_ids,
                                 const float* learning_rate, const T* scale_by_ptr,
                                 const T* down_scale_by_ptr, const int64_t* skip_if,
                                 const G* model_diff, const T* unique_values,
                                 T* updated_unique_values) {
  if (skip_if != nullptr && *skip_if != 0) {
    const int64_t n = *num_unique_ids * line_size;
    CUDA_1D_KERNEL_LOOP(i, n) {
      // The n is the unique_values elem_cnt, so not need to use GetAdamOffset.
      updated_unique_values[i] = unique_values[i];
    }
  } else {
    if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
    if (down_scale_by_ptr != nullptr) { scale /= *down_scale_by_ptr; }
    float bias_correction1_val = 1.0;
    float bias_correction2_val = 1.0;
    if (bias_correction1_ptr != nullptr) { bias_correction1_val = *bias_correction1_ptr; }
    if (bias_correction2_ptr != nullptr) { bias_correction2_val = *bias_correction2_ptr; }
    float learning_rate_val = *learning_rate;
    const int64_t n = *num_unique_ids * embedding_size;
    // The n is model_diff elem_cnt.
    CUDA_1D_KERNEL_LOOP(i, n) {
      int64_t model_offset;
      int64_t m_offset;
      int64_t v_offset;
      GetAdamOffset(line_size, embedding_size, i, &model_offset, &m_offset, &v_offset);
      updated_unique_values[model_offset] = unique_values[model_offset];
      updated_unique_values[m_offset] = unique_values[m_offset];
      updated_unique_values[v_offset] = unique_values[v_offset];
      AdamUpdateFunctor<T, G>()(model_diff + i, updated_unique_values + model_offset,
                                updated_unique_values + m_offset, updated_unique_values + v_offset,
                                nullptr, scale, l1, l2, beta1, beta2, epsilon, weight_decay, false,
                                bias_correction1_val, bias_correction2_val, learning_rate_val);
    }
  }
}

template<typename T, typename G, typename IDX>
__global__ void AdagradUpdateKernel(const int64_t line_size, const int64_t embedding_size, T scale,
                                    float l1, float l2, float weight_decay, float lr_decay,
                                    float epsilon, const IDX* num_unique_ids,
                                    const float* learning_rate, const int64_t* train_step_ptr,
                                    const T* scale_by_ptr, const T* down_scale_by_ptr,
                                    const int64_t* skip_if, const G* model_diff,
                                    const T* unique_values, T* updated_unique_values) {
  if (skip_if != nullptr && *skip_if != 0) {
    const int64_t n = *num_unique_ids * line_size;
    CUDA_1D_KERNEL_LOOP(i, n) { updated_unique_values[i] = unique_values[i]; }
  } else {
    int64_t train_step = *train_step_ptr + 1;
    if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
    if (down_scale_by_ptr != nullptr) { scale /= *down_scale_by_ptr; }
    float learning_rate_val = *learning_rate;
    learning_rate_val = learning_rate_val / (1 + (train_step - 1) * lr_decay);
    const int64_t n = *num_unique_ids * embedding_size;
    CUDA_1D_KERNEL_LOOP(i, n) {
      int64_t model_offset;
      int64_t sum_offset;
      GetMomentumOffset(line_size, embedding_size, i, &model_offset, &sum_offset);
      updated_unique_values[model_offset] = unique_values[model_offset];
      updated_unique_values[sum_offset] = unique_values[sum_offset];
      AdagradUpdateFunctor<T, G>()(model_diff + i, updated_unique_values + model_offset,
                                   updated_unique_values + sum_offset, scale, l1, l2, epsilon,
                                   weight_decay, learning_rate_val);
    }
  }
}

__device__ void GetFtrlOffset(const int32_t line_size, const int32_t embedding_size,
                              int64_t model_diff_offset, int64_t* model_offset,
                              int64_t* accumulate_offset, int64_t* z_offset) {
  const int32_t row = model_diff_offset / embedding_size;
  const int32_t col = model_diff_offset - row * embedding_size;
  *model_offset = row * line_size + col;
  *accumulate_offset = *model_offset + embedding_size;
  *z_offset = *model_offset + 2 * embedding_size;
}

template<typename T, typename G, typename IDX>
__global__ void FtrlUpdateKernel(const int32_t line_size, const int32_t embedding_size, T scale,
                                 float l1, float l2, float weight_decay, float lr_power,
                                 float lambda1, float lambda2, float beta,
                                 const IDX* num_unique_ids, const float* learning_rate,
                                 const T* down_scale_by_ptr, const int64_t* skip_if,
                                 const G* model_diff, const T* unique_values,
                                 T* updated_unique_values) {
  if (skip_if != nullptr && *skip_if != 0) {
    const int64_t n = *num_unique_ids * line_size;
    CUDA_1D_KERNEL_LOOP(i, n) { updated_unique_values[i] = unique_values[i]; }
  } else {
    if (down_scale_by_ptr != nullptr) { scale /= *down_scale_by_ptr; }
    float learning_rate_val = *learning_rate;
    const int64_t n = *num_unique_ids * embedding_size;
    CUDA_1D_KERNEL_LOOP(i, n) {
      int64_t model_offset;
      int64_t accumulate_offset;
      int64_t z_offset;
      GetFtrlOffset(line_size, embedding_size, i, &model_offset, &accumulate_offset, &z_offset);
      updated_unique_values[model_offset] = unique_values[model_offset];
      updated_unique_values[accumulate_offset] = unique_values[accumulate_offset];
      updated_unique_values[z_offset] = unique_values[z_offset];
      FtrlUpdateFunctor<T, G>()(model_diff + i, updated_unique_values + model_offset,
                                updated_unique_values + accumulate_offset,
                                updated_unique_values + z_offset, scale, l1, l2, lr_power, lambda1,
                                lambda2, beta, weight_decay, learning_rate_val);
    }
  }
}

}  // namespace

template<typename T, typename G, typename IDX>
class SgdEmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  SgdEmbeddingUpdateKernel() = default;
  ~SgdEmbeddingUpdateKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_embeddings = ctx->Tensor4ArgNameAndIndex("unique_embeddings", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    user_op::Tensor* updated_unique_embeddings =
        ctx->Tensor4ArgNameAndIndex("updated_unique_embeddings", 0);
    CHECK_EQ(unique_embeddings->shape().NumAxes(), 2);
    CHECK_EQ(embedding_grad->shape().NumAxes(), 2);
    const int64_t line_size = unique_embeddings->shape().At(1);
    const int64_t embedding_size = embedding_grad->shape().At(1);
    CHECK_EQ(line_size, embedding_size);
    const auto scale = ctx->Attr<double>("scale");
    const float l1 = ctx->Attr<float>("l1");
    const float l2 = ctx->Attr<float>("l2");
    const auto weight_decay = ctx->Attr<float>("weight_decay");
    const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
    const float* learning_rate_ptr = learning_rate->dptr<float>();
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->data_type(), unique_embeddings->data_type());
      CHECK_EQ(scale_by_tensor->shape().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const T* down_scale_by_ptr = nullptr;
    if (ctx->has_input("down_scale_by_tensor", 0)) {
      const user_op::Tensor* down_scale_by_tensor =
          ctx->Tensor4ArgNameAndIndex("down_scale_by_tensor", 0);
      CHECK_EQ(down_scale_by_tensor->data_type(), unique_embeddings->data_type());
      CHECK_EQ(down_scale_by_tensor->shape().elem_cnt(), 1);
      down_scale_by_ptr = down_scale_by_tensor->dptr<T>();
    }
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }
    // update kernel
    SGDUpdateKernel<T, G, IDX>
        <<<BlocksNum4ThreadsNum(embedding_grad->shape().elem_cnt()), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            embedding_size, scale, l1, l2, weight_decay,
            reinterpret_cast<const IDX*>(num_unique_ids->dptr()), learning_rate_ptr, scale_by_ptr,
            down_scale_by_ptr, skip_if_ptr, embedding_grad->dptr<G>(), unique_embeddings->dptr<T>(),
            updated_unique_embeddings->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define IDX_DATA_TYPE_SEQ                           \
  OF_PP_MAKE_TUPLE_SEQ(uint32_t, DataType::kUInt32) \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32)

#define REGISTER_CUDA_SGD_EMBEDDING_UPDATE_KERNEL(t_dtype_pair, g_type_pair, idx_dtype_pair)      \
  REGISTER_USER_KERNEL("sgd_embedding_update")                                                    \
      .SetCreateFn<                                                                               \
          SgdEmbeddingUpdateKernel<OF_PP_PAIR_FIRST(t_dtype_pair), OF_PP_PAIR_FIRST(g_type_pair), \
                                   OF_PP_PAIR_FIRST(idx_dtype_pair)>>()                           \
      .SetIsMatchedHob(                                                                           \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                         \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))     \
          && (user_op::HobDataType("embedding_grad", 0) == OF_PP_PAIR_SECOND(g_type_pair))        \
          && (user_op::HobDataType("unique_embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair)));

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_SGD_EMBEDDING_UPDATE_KERNEL, FLOATING_DATA_TYPE_SEQ,
                                 FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ, IDX_DATA_TYPE_SEQ)

template<typename T, typename G, typename IDX>
class MomentumEmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  MomentumEmbeddingUpdateKernel() = default;
  ~MomentumEmbeddingUpdateKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_embeddings = ctx->Tensor4ArgNameAndIndex("unique_embeddings", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    user_op::Tensor* updated_unique_embeddings =
        ctx->Tensor4ArgNameAndIndex("updated_unique_embeddings", 0);
    CHECK_EQ(unique_embeddings->shape().NumAxes(), 2);
    CHECK_EQ(embedding_grad->shape().NumAxes(), 2);
    const int64_t num_keys = unique_embeddings->shape().At(0);
    const int64_t line_size = unique_embeddings->shape().At(1);
    const int64_t embedding_size = embedding_grad->shape().At(1);
    CHECK_EQ(line_size, embedding_size * 2);
    const float l1 = ctx->Attr<float>("l1");
    const float l2 = ctx->Attr<float>("l2");
    const auto weight_decay = ctx->Attr<float>("weight_decay");
    const auto beta = ctx->Attr<float>("beta");
    const auto scale = ctx->Attr<double>("scale");
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->data_type(), unique_embeddings->data_type());
      CHECK_EQ(scale_by_tensor->shape().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const T* down_scale_by_ptr = nullptr;
    if (ctx->has_input("down_scale_by_tensor", 0)) {
      const user_op::Tensor* down_scale_by_tensor =
          ctx->Tensor4ArgNameAndIndex("down_scale_by_tensor", 0);
      CHECK_EQ(down_scale_by_tensor->data_type(), unique_embeddings->data_type());
      CHECK_EQ(down_scale_by_tensor->shape().elem_cnt(), 1);
      down_scale_by_ptr = down_scale_by_tensor->dptr<T>();
    }
    const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
    const float* learning_rate_ptr = learning_rate->dptr<float>();
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }
    // update kernel
    MomentumUpdateKernel<T, G, IDX>
        <<<BlocksNum4ThreadsNum(embedding_grad->shape().elem_cnt()), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            line_size, embedding_size, scale, l1, l2, weight_decay, beta,
            reinterpret_cast<const IDX*>(num_unique_ids->dptr()), learning_rate_ptr, scale_by_ptr,
            down_scale_by_ptr, skip_if_ptr, embedding_grad->dptr<G>(), unique_embeddings->dptr<T>(),
            updated_unique_embeddings->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_MOMENTUM_EMBEDDING_UPDATE_KERNEL(t_dtype_pair, g_type_pair, idx_dtype_pair) \
  REGISTER_USER_KERNEL("momentum_embedding_update")                                               \
      .SetCreateFn<MomentumEmbeddingUpdateKernel<OF_PP_PAIR_FIRST(t_dtype_pair),                  \
                                                 OF_PP_PAIR_FIRST(g_type_pair),                   \
                                                 OF_PP_PAIR_FIRST(idx_dtype_pair)>>()             \
      .SetIsMatchedHob(                                                                           \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                         \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))     \
          && (user_op::HobDataType("embedding_grad", 0) == OF_PP_PAIR_SECOND(g_type_pair))        \
          && (user_op::HobDataType("unique_embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair)));

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_MOMENTUM_EMBEDDING_UPDATE_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ,
                                 IDX_DATA_TYPE_SEQ)

template<typename T, typename G, typename IDX>
class AdamEmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  AdamEmbeddingUpdateKernel() = default;
  ~AdamEmbeddingUpdateKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_embeddings = ctx->Tensor4ArgNameAndIndex("unique_embeddings", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    user_op::Tensor* updated_unique_embeddings =
        ctx->Tensor4ArgNameAndIndex("updated_unique_embeddings", 0);
    CHECK_EQ(unique_embeddings->shape().NumAxes(), 2);
    CHECK_EQ(embedding_grad->shape().NumAxes(), 2);
    const int64_t num_keys = unique_embeddings->shape().At(0);
    const int64_t line_size = unique_embeddings->shape().At(1);
    const int64_t embedding_size = embedding_grad->shape().At(1);
    CHECK_EQ(line_size, embedding_size * 3);

    const float l1 = ctx->Attr<float>("l1");
    const float l2 = ctx->Attr<float>("l2");
    const auto weight_decay = ctx->Attr<float>("weight_decay");
    const auto beta1 = ctx->Attr<float>("beta1");
    const auto beta2 = ctx->Attr<float>("beta2");
    const auto epsilon = ctx->Attr<float>("epsilon");
    const bool do_bias_correction = ctx->Attr<bool>("do_bias_correction");
    const auto scale = ctx->Attr<double>("scale");
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->data_type(), unique_embeddings->data_type());
      CHECK_EQ(scale_by_tensor->shape().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const T* down_scale_by_ptr = nullptr;
    if (ctx->has_input("down_scale_by_tensor", 0)) {
      const user_op::Tensor* down_scale_by_tensor =
          ctx->Tensor4ArgNameAndIndex("down_scale_by_tensor", 0);
      CHECK_EQ(down_scale_by_tensor->data_type(), unique_embeddings->data_type());
      CHECK_EQ(down_scale_by_tensor->shape().elem_cnt(), 1);
      down_scale_by_ptr = down_scale_by_tensor->dptr<T>();
    }
    const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
    const float* learning_rate_ptr = learning_rate->dptr<float>();
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }
    const float* bias_correction1_ptr = nullptr;
    if (ctx->has_input("bias_correction1", 0)) {
      bias_correction1_ptr = ctx->Tensor4ArgNameAndIndex("bias_correction1", 0)->dptr<float>();
    }
    const float* bias_correction2_ptr = nullptr;
    if (ctx->has_input("bias_correction2", 0)) {
      bias_correction2_ptr = ctx->Tensor4ArgNameAndIndex("bias_correction2", 0)->dptr<float>();
    }
    // update kernel
    AdamUpdateKernel<T, G, IDX>
        <<<BlocksNum4ThreadsNum(embedding_grad->shape().elem_cnt()), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            line_size, embedding_size, static_cast<T>(scale), l1, l2, weight_decay, beta1, beta2,
            epsilon, bias_correction1_ptr, bias_correction2_ptr,
            reinterpret_cast<const IDX*>(num_unique_ids->dptr()), learning_rate_ptr, scale_by_ptr,
            down_scale_by_ptr, skip_if_ptr, embedding_grad->dptr<G>(), unique_embeddings->dptr<T>(),
            updated_unique_embeddings->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_ADAM_EMBEDDING_UPDATE_KERNEL(t_dtype_pair, g_type_pair, idx_dtype_pair)      \
  REGISTER_USER_KERNEL("adam_embedding_update")                                                    \
      .SetCreateFn<                                                                                \
          AdamEmbeddingUpdateKernel<OF_PP_PAIR_FIRST(t_dtype_pair), OF_PP_PAIR_FIRST(g_type_pair), \
                                    OF_PP_PAIR_FIRST(idx_dtype_pair)>>()                           \
      .SetIsMatchedHob(                                                                            \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                          \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))      \
          && (user_op::HobDataType("embedding_grad", 0) == OF_PP_PAIR_SECOND(g_type_pair))         \
          && (user_op::HobDataType("unique_embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair)));

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_ADAM_EMBEDDING_UPDATE_KERNEL, FLOATING_DATA_TYPE_SEQ,
                                 FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ, IDX_DATA_TYPE_SEQ)

template<typename T, typename G, typename IDX>
class AdagradEmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  AdagradEmbeddingUpdateKernel() = default;
  ~AdagradEmbeddingUpdateKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_embeddings = ctx->Tensor4ArgNameAndIndex("unique_embeddings", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    user_op::Tensor* updated_unique_embeddings =
        ctx->Tensor4ArgNameAndIndex("updated_unique_embeddings", 0);
    CHECK_EQ(unique_embeddings->shape().NumAxes(), 2);
    CHECK_EQ(embedding_grad->shape().NumAxes(), 2);
    const int64_t num_keys = unique_embeddings->shape().At(0);
    const int64_t line_size = unique_embeddings->shape().At(1);
    const int64_t embedding_size = embedding_grad->shape().At(1);
    CHECK_EQ(line_size, embedding_size * 2);

    const float l1 = ctx->Attr<float>("l1");
    const float l2 = ctx->Attr<float>("l2");
    const auto weight_decay = ctx->Attr<float>("weight_decay");
    const auto lr_decay = ctx->Attr<float>("lr_decay");
    const auto epsilon = ctx->Attr<float>("epsilon");
    const auto scale = ctx->Attr<double>("scale");
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->data_type(), unique_embeddings->data_type());
      CHECK_EQ(scale_by_tensor->shape().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const T* down_scale_by_ptr = nullptr;
    if (ctx->has_input("down_scale_by_tensor", 0)) {
      const user_op::Tensor* down_scale_by_tensor =
          ctx->Tensor4ArgNameAndIndex("down_scale_by_tensor", 0);
      CHECK_EQ(down_scale_by_tensor->data_type(), unique_embeddings->data_type());
      CHECK_EQ(down_scale_by_tensor->shape().elem_cnt(), 1);
      down_scale_by_ptr = down_scale_by_tensor->dptr<T>();
    }
    const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
    const float* learning_rate_ptr = learning_rate->dptr<float>();
    const int64_t* train_step_ptr = ctx->Tensor4ArgNameAndIndex("train_step", 0)->dptr<int64_t>();
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }
    // update kernel
    AdagradUpdateKernel<T, G, IDX>
        <<<BlocksNum4ThreadsNum(embedding_grad->shape().elem_cnt()), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            line_size, embedding_size, static_cast<T>(scale), l1, l2, weight_decay, lr_decay,
            epsilon, reinterpret_cast<const IDX*>(num_unique_ids->dptr()), learning_rate_ptr,
            train_step_ptr, scale_by_ptr, down_scale_by_ptr, skip_if_ptr, embedding_grad->dptr<G>(),
            unique_embeddings->dptr<T>(), updated_unique_embeddings->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_ADAGRAD_EMBEDDING_UPDATE_KERNEL(t_dtype_pair, g_type_pair, idx_dtype_pair) \
  REGISTER_USER_KERNEL("adagrad_embedding_update")                                               \
      .SetCreateFn<AdagradEmbeddingUpdateKernel<OF_PP_PAIR_FIRST(t_dtype_pair),                  \
                                                OF_PP_PAIR_FIRST(g_type_pair),                   \
                                                OF_PP_PAIR_FIRST(idx_dtype_pair)>>()             \
      .SetIsMatchedHob(                                                                          \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                        \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))    \
          && (user_op::HobDataType("embedding_grad", 0) == OF_PP_PAIR_SECOND(g_type_pair))       \
          && (user_op::HobDataType("unique_embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair)));

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_ADAGRAD_EMBEDDING_UPDATE_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ,
                                 IDX_DATA_TYPE_SEQ)

template<typename T, typename G, typename IDX>
class FtrlEmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  FtrlEmbeddingUpdateKernel() = default;
  ~FtrlEmbeddingUpdateKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_embeddings = ctx->Tensor4ArgNameAndIndex("unique_embeddings", 0);
    const user_op::Tensor* embedding_grad = ctx->Tensor4ArgNameAndIndex("embedding_grad", 0);
    user_op::Tensor* updated_unique_embeddings =
        ctx->Tensor4ArgNameAndIndex("updated_unique_embeddings", 0);
    CHECK_EQ(unique_embeddings->shape().NumAxes(), 2)
        << "The NumAxes of unique_embedding should be equal to 2. ";
    CHECK_EQ(embedding_grad->shape().NumAxes(), 2)
        << "The NumAxes of embedding_grad should be equal to 2. ";
    const int64_t num_keys = unique_embeddings->shape().At(0);
    const int64_t line_size = unique_embeddings->shape().At(1);
    const int64_t embedding_size = embedding_grad->shape().At(1);
    CHECK_EQ(line_size, embedding_size * 3)
        << "The line_size should be equal to 3 x embedding_size. ";
    const float l1 = 0.0;
    const float l2 = 0.0;
    const float weight_decay = ctx->Attr<float>("weight_decay");
    // TODO(zhengzekang): Undefined behavior for ftrl optimizer with weight_decay in `abs(new_z_val)
    // < lambda1` condition.
    CHECK_EQ(weight_decay, static_cast<float>(0.0))
        << "Currently not support for setting weight decay. ";
    const float lr_power = ctx->Attr<float>("lr_power");
    const float lambda1 = ctx->Attr<float>("lambda1");
    const float lambda2 = ctx->Attr<float>("lambda2");
    const float beta = ctx->Attr<float>("beta");
    const double scale = ctx->Attr<double>("scale");
    const T* down_scale_by_ptr = nullptr;
    if (ctx->has_input("down_scale_by_tensor", 0)) {
      const user_op::Tensor* down_scale_by_tensor =
          ctx->Tensor4ArgNameAndIndex("down_scale_by_tensor", 0);
      CHECK_EQ(down_scale_by_tensor->data_type(), unique_embeddings->data_type());
      CHECK_EQ(down_scale_by_tensor->shape().elem_cnt(), 1);
      down_scale_by_ptr = down_scale_by_tensor->dptr<T>();
    }
    const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
    const float* learning_rate_ptr = learning_rate->dptr<float>();
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }
    // update kernel
    FtrlUpdateKernel<T, G, IDX>
        <<<BlocksNum4ThreadsNum(embedding_grad->shape().elem_cnt()), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            line_size, embedding_size, static_cast<T>(scale), l1, l2, weight_decay, lr_power,
            lambda1, lambda2, beta, reinterpret_cast<const IDX*>(num_unique_ids->dptr()),
            learning_rate_ptr, down_scale_by_ptr, skip_if_ptr, embedding_grad->dptr<G>(),
            unique_embeddings->dptr<T>(), updated_unique_embeddings->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};
#define REGISTER_CUDA_FTRL_EMBEDDING_UPDATE_KERNEL(t_dtype_pair, g_type_pair, idx_dtype_pair)      \
  REGISTER_USER_KERNEL("ftrl_embedding_update")                                                    \
      .SetCreateFn<                                                                                \
          FtrlEmbeddingUpdateKernel<OF_PP_PAIR_FIRST(t_dtype_pair), OF_PP_PAIR_FIRST(g_type_pair), \
                                    OF_PP_PAIR_FIRST(idx_dtype_pair)>>()                           \
      .SetIsMatchedHob(                                                                            \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                          \
          && (user_op::HobDataType("num_unique_ids", 0) == OF_PP_PAIR_SECOND(idx_dtype_pair))      \
          && (user_op::HobDataType("embedding_grad", 0) == OF_PP_PAIR_SECOND(g_type_pair))         \
          && (user_op::HobDataType("unique_embeddings", 0) == OF_PP_PAIR_SECOND(t_dtype_pair)));
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CUDA_FTRL_EMBEDDING_UPDATE_KERNEL, FLOATING_DATA_TYPE_SEQ,
                                 FLOATING_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ, IDX_DATA_TYPE_SEQ)

}  // namespace oneflow
