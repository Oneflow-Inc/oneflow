#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/kernel/util/cuda_half_util.h"
#include "oneflow/core/cuda/elementwise.cuh"

namespace oneflow {

namespace user_op {

template<typename T>
struct HardswishFunctor {
  OF_DEVICE_FUNC T operator()(T x) const {
    if (x <= static_cast<T>(-3)) {
      return static_cast<T>(0);
    } else if (x >= static_cast<T>(3)) {
      return x;
    } else {
      return (x * (x + static_cast<T>(3))) / static_cast<T>(6);
    }
  }
};

template<typename T>
struct HardswishGradFunctor {
  OF_DEVICE_FUNC T operator()(T x, T dy) const {
    if (x <= static_cast<T>(-3)) {
      return static_cast<T>(0);
    } else if (x >= static_cast<T>(3)) {
      return dy;
    } else {
      return ((x / static_cast<T>(3)) + static_cast<T>(0.5)) * dy;
    }
  }
};

template<>
struct HardswishFunctor<half> {
  HardswishFunctor<float> float_functor;
  OF_DEVICE_FUNC half operator()(half x) const {
    return __float2half(float_functor(__half2float(x)));
  }
};

template<>
struct HardswishGradFunctor<half> {
  HardswishGradFunctor<float> float_functor;
  OF_DEVICE_FUNC half operator()(half x, half dy) const {
    return __float2half(float_functor(__half2float(x), __half2float(dy)));
  }
};

template<DeviceType device_type, typename T>
class GpuHardswishKernel final : public OpKernel {
 public:
  GpuHardswishKernel() = default;
  ~GpuHardswishKernel() = default;

 private:
  void Compute(KernelComputeContext* ctx) const override {
    const Tensor* in_tensor = ctx->Tensor4ArgNameAndIndex("in", 0);
    Tensor* out_tensor = ctx->Tensor4ArgNameAndIndex("out", 0);
    const T* in_ptr = in_tensor->dptr<T>();
    T* out_ptr = out_tensor->mut_dptr<T>();

    const int32_t elem_cnt = in_tensor->shape().elem_cnt();
    OF_CUDA_CHECK((oneflow::cuda::elementwise::Unary(HardswishFunctor<T>(), elem_cnt, out_ptr,
                                                     in_ptr, ctx->device_ctx()->hip_stream())));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GPU_HARDSWISH_KERNEL(device, dtype)    \
  REGISTER_USER_KERNEL("hardswish")                     \
      .SetCreateFn<GpuHardswishKernel<device, dtype>>() \
      .SetIsMatchedHob((HobDeviceTag() == device)       \
                       & (HobDataType("out", 0) == GetDataType<dtype>::value));

REGISTER_GPU_HARDSWISH_KERNEL(DeviceType::kGPU, half)
REGISTER_GPU_HARDSWISH_KERNEL(DeviceType::kGPU, float)
REGISTER_GPU_HARDSWISH_KERNEL(DeviceType::kGPU, double)

template<DeviceType device_type, typename T>
class GpuHardswishGradKernel final : public OpKernel {
 public:
  GpuHardswishGradKernel() = default;
  ~GpuHardswishGradKernel() = default;

 private:
  void Compute(KernelComputeContext* ctx) const override {
    const Tensor* x_tensor = ctx->Tensor4ArgNameAndIndex("x", 0);
    const Tensor* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    Tensor* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const T* x_ptr = x_tensor->dptr<T>();
    const T* dy_ptr = dy_tensor->dptr<T>();
    T* dx_ptr = dx_tensor->mut_dptr<T>();

    const int32_t elem_cnt = x_tensor->shape().elem_cnt();
    OF_CUDA_CHECK(
        (oneflow::cuda::elementwise::Binary(HardswishGradFunctor<T>(), elem_cnt, dx_ptr, x_ptr,
                                            dy_ptr, ctx->device_ctx()->hip_stream())));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GPU_HARDSWISH_BACKWARD_KERNEL(device, dtype) \
  REGISTER_USER_KERNEL("hardswish_grad")                      \
      .SetCreateFn<GpuHardswishGradKernel<device, dtype>>()   \
      .SetIsMatchedHob((HobDeviceTag() == device)             \
                       & (HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_GPU_HARDSWISH_BACKWARD_KERNEL(DeviceType::kGPU, half)
REGISTER_GPU_HARDSWISH_BACKWARD_KERNEL(DeviceType::kGPU, float)
REGISTER_GPU_HARDSWISH_BACKWARD_KERNEL(DeviceType::kGPU, double)

}  // namespace user_op

}  // namespace oneflow
