#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/range_kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void RangeForwardGpuKernel(const int start, const int delta, const int range_shape,
                                      T* out) {
  // Use Loop to set the value
  CUDA_1D_KERNEL_LOOP(i, range_shape) { out[i] = start + i * delta; }
}

}  // namespace

template<typename T>
struct RangeKernelUtil<DeviceType::kGPU, T> {
  static void Range(DeviceCtx* ctx, const int start, const int delta, const int range_shape,
                    T* out) {
    // Run cuda range forward kernel
    // The thread num is set as range_shape
    RUN_CUDA_KERNEL(RangeForwardGpuKernel, ctx, range_shape, start, delta, range_shape, out);
  }
};

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_RANGE_FUNCTOR, (DeviceType::kGPU),
                                 ARITHMETIC_DATA_TYPE_SEQ)

}  // namespace oneflow
