#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/clip_by_value_kernel.h"
#include "oneflow/core/device/cuda_util.h"

namespace oneflow {

namespace {

template<typename T, typename F>
__global__ void CudaClipForward(F clip_func, int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = clip_func(x[i]); }
}

template<typename T, typename F>
__global__ void CudaClipBackward(F clip_func, int64_t n, const T* x, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = clip_func(x[i], dy[i]); }
}

}  // namespace

template<typename T>
struct ClipKernelUtil<DeviceType::kGPU, T> {
  template<typename F>
  static void Forward(DeviceCtx* ctx, F clip_func, const int64_t n, const T* x, T* y) {
    RUN_CUDA_KERNEL((CudaClipForward<T, F>), ctx, n, clip_func, n, x, y);
  }

  template<typename F>
  static void Backward(DeviceCtx* ctx, F clip_func, const int64_t n, const T* x, const T* dy,
                       T* dx) {
    RUN_CUDA_KERNEL((CudaClipBackward<T, F>), ctx, n, clip_func, n, x, dy, dx);
  }
};

#define INITIATE_CLIP_KERNEL_UTIL_GPU(dtype, dtype_v)                                          \
  template struct ClipKernelUtil<DeviceType::kGPU, dtype>;                                     \
  template void ClipKernelUtil<DeviceType::kGPU, dtype>::Forward(                              \
      DeviceCtx*, ClipByMinFunctor<dtype>, const int64_t n, const dtype*, dtype*);             \
  template void ClipKernelUtil<DeviceType::kGPU, dtype>::Forward(                              \
      DeviceCtx*, ClipByMaxFunctor<dtype>, const int64_t n, const dtype*, dtype*);             \
  template void ClipKernelUtil<DeviceType::kGPU, dtype>::Forward(                              \
      DeviceCtx*, ClipByMinMaxFunctor<dtype>, const int64_t n, const dtype*, dtype*);          \
  template void ClipKernelUtil<DeviceType::kGPU, dtype>::Backward(                             \
      DeviceCtx*, ClipByMinGradFunctor<dtype>, const int64_t n, const dtype*, const dtype*,    \
      dtype*);                                                                                 \
  template void ClipKernelUtil<DeviceType::kGPU, dtype>::Backward(                             \
      DeviceCtx*, ClipByMaxGradFunctor<dtype>, const int64_t n, const dtype*, const dtype*,    \
      dtype*);                                                                                 \
  template void ClipKernelUtil<DeviceType::kGPU, dtype>::Backward(                             \
      DeviceCtx*, ClipByMinMaxGradFunctor<dtype>, const int64_t n, const dtype*, const dtype*, \
      dtype*);

OF_PP_FOR_EACH_TUPLE(INITIATE_CLIP_KERNEL_UTIL_GPU, ARITHMETIC_DATA_TYPE_SEQ)

}  // namespace oneflow
