#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/cuda/softmax.cuh"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/user/kernels/fused_softmax.cuh"

namespace oneflow {
namespace cuda {

namespace {

template<typename IndexType, size_t NDIM>
struct BroadcastMapper {
  using index_type = IndexType;
  IndexType src_dims[NDIM] = {0};
  IndexType dst_dims[NDIM] = {0};

  template<typename DimType>
  BroadcastMapper(const DimType* arg_src_dims, const DimType* arg_dst_dims) {
    for (size_t i = 0; i < NDIM; ++i) { src_dims[i] = arg_src_dims[i]; }
    for (size_t i = 0; i < NDIM; ++i) { dst_dims[i] = arg_dst_dims[i]; }
  }

  __device__ IndexType map(IndexType src) const {
    NdIndexOffsetHelper<IndexType, NDIM> src_index_helper(src_dims);
    NdIndexOffsetHelper<IndexType, NDIM> dst_index_helper(dst_dims);
    IndexType src_index[NDIM];
    IndexType dst_index[NDIM];
    src_index_helper.OffsetToNdIndex(src, src_index);
#pragma unroll
    for (int dim = 0; dim < NDIM; ++dim) {
      if (dst_dims[dim] == 1) {
        dst_index[dim] = 0;
      } else {
        dst_index[dim] = src_index[dim];
      }
    }
    return dst_index_helper.NdIndexToOffset(dst_index);
  }
};

template<typename IndexType>
struct ElementwiseMapper {
  using index_type = IndexType;
  ElementwiseMapper() {}
  __device__ IndexType map(IndexType index) const { return index; }
};

template<typename SRC, typename DST, typename MASK, typename BiasMapper, typename MaskMapper>
struct BiasAddScaleMaskLoad {
  static_assert(
      std::is_same<typename BiasMapper::index_type, typename MaskMapper::index_type>::value, "");
  using IndexType = typename BiasMapper::index_type;
  const SRC* src;
  const SRC* bias;
  const MASK* mask;
  const DST fill;
  const DST scale;
  const IndexType row_size;
  const BiasMapper bias_mapper;
  const MaskMapper mask_mapper;

  BiasAddScaleMaskLoad(const SRC* src, const SRC* bias, const MASK* mask, const DST fill,
                       const DST scale, const IndexType row_size, const BiasMapper bias_mapper,
                       const MaskMapper mask_mapper)
      : src(src),
        bias(bias),
        mask(mask),
        fill(fill),
        scale(scale),
        row_size(row_size),
        bias_mapper(bias_mapper),
        mask_mapper(mask_mapper) {}

  template<int N>
  __device__ void load(DST* dst, IndexType row, IndexType col) {
    softmax::Pack<SRC, N> src_pack;
    softmax::Pack<SRC, N> bias_pack;
    softmax::Pack<MASK, N> mask_pack;
    const IndexType offset = row * row_size + col;
    const IndexType bias_offset = bias_mapper.map(offset);
    const IndexType mask_offset = mask_mapper.map(offset);
    src_pack.storage = *(reinterpret_cast<const softmax::PackType<SRC, N>*>(src) + offset / N);
    bias_pack.storage =
        *(reinterpret_cast<const softmax::PackType<SRC, N>*>(bias) + bias_offset / N);
    mask_pack.storage =
        *(reinterpret_cast<const softmax::PackType<MASK, N>*>(mask) + mask_offset / N);
#pragma unroll
    for (int i = 0; i < N; ++i) {
      if (mask_pack.elem[i] == 0) {
        dst[i] = fill;
      } else {
        dst[i] = static_cast<DST>(src_pack.elem[i] + bias_pack.elem[i]) * scale;
      }
    }
  }
};

template<typename T, typename MASK>
void DispatchForward(hipStream_t stream, const user_op::Tensor* x, const user_op::Tensor* bias,
                     const user_op::Tensor* mask, const user_op::Tensor* dropout_mask,
                     const float mask_fill, const float scale, const float dropout_scale,
                     user_op::Tensor* y, user_op::Tensor* softmax_y) {
  using ComputeType = typename softmax::DefaultComputeType<T>::type;
  using IndexType = int32_t;
  constexpr int kMaxNDim = 5;

  const auto& x_shape = x->shape_view();
  CHECK_GE(x_shape.size(), 2);
  // the last dim is softmax dim which is considered as col
  int64_t ncol = x_shape[x_shape.size() - 1];
  int64_t nrow = x_shape.elem_cnt() / ncol;
  fused_softmax::DropoutStore<ComputeType, T> store(
      y->mut_dptr<T>(), softmax_y->mut_dptr<T>(), dropout_mask->dptr<bool>(), ncol, dropout_scale);

  size_t bias_sndim = 0;
  int64_t bias_x_sdims[kMaxNDim];
  int64_t bias_sdims[kMaxNDim];
  const auto& bias_shape = bias->shape_view();
  fused_softmax::SimplifyBroadcastDims(x_shape.size(), x_shape.ptr(), bias_shape.size(),
                                       bias_shape.ptr(), &bias_sndim, bias_x_sdims, bias_sdims);
  size_t mask_sndim = 0;
  int64_t mask_x_sdims[kMaxNDim];
  int64_t mask_sdims[kMaxNDim];
  const auto& mask_shape = mask->shape_view();
  fused_softmax::SimplifyBroadcastDims(x_shape.size(), x_shape.ptr(), mask_shape.size(),
                                       mask_shape.ptr(), &mask_sndim, mask_x_sdims, mask_sdims);

#define DISPATCH_BIAS_ADD_SCALE_MASK_SOFTMAX(bias_mapper, mask_mapper)                           \
  BiasAddScaleMaskLoad<T, ComputeType, MASK, decltype(bias_mapper), decltype(mask_mapper)> load( \
      x->dptr<T>(), bias->dptr<T>(), mask->dptr<MASK>(), mask_fill, scale, ncol, bias_mapper,    \
      mask_mapper);                                                                              \
  OF_CUDA_CHECK((cuda::softmax::DispatchSoftmax<decltype(load), decltype(store), ComputeType>(   \
      stream, load, store, nrow, ncol)))

  if (bias_sndim == 1 && mask_sndim == 1) {
    // bias elementwise
    // mask elementwise
    ElementwiseMapper<IndexType> bias_mapper;
    ElementwiseMapper<IndexType> mask_mapper;
    DISPATCH_BIAS_ADD_SCALE_MASK_SOFTMAX(bias_mapper, mask_mapper);
  } else if (bias_sndim == 1 && mask_sndim == 2) {
    // bias elementwise
    // mask broadcast: (M, 1) -> (M, N) or (1, N) -> (M, N)
    ElementwiseMapper<IndexType> bias_mapper;
    BroadcastMapper<IndexType, 2> mask_mapper(mask_x_sdims, mask_sdims);
    DISPATCH_BIAS_ADD_SCALE_MASK_SOFTMAX(bias_mapper, mask_mapper);
  } else if (bias_sndim == 1 && mask_sndim == 3) {
    // bias elementwise
    // mask broadcast: (M, 1, N) -> (M, K, N)
    ElementwiseMapper<IndexType> bias_mapper;
    BroadcastMapper<IndexType, 3> mask_mapper(mask_x_sdims, mask_sdims);
    DISPATCH_BIAS_ADD_SCALE_MASK_SOFTMAX(bias_mapper, mask_mapper);
  } else if (bias_sndim == 2 && mask_sndim == 1) {
    // bias broadcast: (M, 1) -> (M, N) or (1, N) -> (M, N)
    // mask elementwise
    BroadcastMapper<IndexType, 2> bias_mapper(bias_x_sdims, bias_sdims);
    ElementwiseMapper<IndexType> mask_mapper;
    DISPATCH_BIAS_ADD_SCALE_MASK_SOFTMAX(bias_mapper, mask_mapper);
  } else if (bias_sndim == 2 && mask_sndim == 2) {
    // bias broadcast: (M, 1) -> (M, N) or (1, N) -> (M, N)
    // mask broadcast: (M, 1) -> (M, N) or (1, N) -> (M, N)
    BroadcastMapper<IndexType, 2> bias_mapper(bias_x_sdims, bias_sdims);
    BroadcastMapper<IndexType, 2> mask_mapper(mask_x_sdims, mask_sdims);
    DISPATCH_BIAS_ADD_SCALE_MASK_SOFTMAX(bias_mapper, mask_mapper);
  } else if (bias_sndim == 2 && mask_sndim == 3) {
    // bias broadcast: (M, 1) -> (M, N) or (1, N) -> (M, N)
    // mask broadcast: (M, 1, N) -> (M, K, N)
    BroadcastMapper<IndexType, 2> bias_mapper(bias_x_sdims, bias_sdims);
    BroadcastMapper<IndexType, 3> mask_mapper(mask_x_sdims, mask_sdims);
    DISPATCH_BIAS_ADD_SCALE_MASK_SOFTMAX(bias_mapper, mask_mapper);
    // not support for now
    // } else if (bias_sndim == 3 && mask_sndim == 1) {
    // } else if (bias_sndim == 3 && mask_sndim == 2) {
    // } else if (bias_sndim == 3 && mask_sndim == 3) {
  } else {
    UNIMPLEMENTED() << ", bias_sndim=" << bias_sndim << ", mask_sndim=" << mask_sndim;
  }

#undef DISPATCH_BIAS_ADD_SCALE_MASK_SOFTMAX
}

template<typename T, typename MASK>
class FusedBiasAddScaleMaskSoftmaxDropoutKernel final : public user_op::OpKernel {
 public:
  FusedBiasAddScaleMaskSoftmaxDropoutKernel() = default;
  ~FusedBiasAddScaleMaskSoftmaxDropoutKernel() override = default;

 private:
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }

  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* bias = ctx->Tensor4ArgNameAndIndex("bias", 0);
    const user_op::Tensor* mask = ctx->Tensor4ArgNameAndIndex("mask", 0);
    const user_op::Tensor* dropout_mask = ctx->Tensor4ArgNameAndIndex("dropout_mask", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    user_op::Tensor* softmax_y = ctx->Tensor4ArgNameAndIndex("softmax_y", 0);

    const float mask_fill = ctx->Attr<float>("mask_fill_value");
    const float scale = ctx->Attr<float>("scale_value");
    const float dropout_scale = ctx->Attr<float>("dropout_scale_value");

    const ShapeView& x_shape = x->shape_view();
    // int32 index computing is much faster than int64
    // TODO: consider using multiple int32 computing to substitute int64 computing
    CHECK_LT(x_shape.elem_cnt(), INT_MAX) << "only support int32 max limits size of elements";
    DispatchForward<T, MASK>(ctx->stream()->As<ep::CudaStream>()->hip_stream(), x, bias, mask,
                             dropout_mask, mask_fill, scale, dropout_scale, y, softmax_y);
  }
};

}  // namespace

#define REGISTER_FUSED_BIAS_ADD_SCALE_MASK_SOFTMAX_DROPOUT_CUDA_KERNEL(dtype, mask_dtype) \
  REGISTER_USER_KERNEL("fused_bias_add_scale_mask_softmax_dropout")                       \
      .SetCreateFn<FusedBiasAddScaleMaskSoftmaxDropoutKernel<dtype, mask_dtype>>()        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                    \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value)     \
                       && (user_op::HobDataType("mask", 0) == GetDataType<mask_dtype>::value));

REGISTER_FUSED_BIAS_ADD_SCALE_MASK_SOFTMAX_DROPOUT_CUDA_KERNEL(float, bool)
REGISTER_FUSED_BIAS_ADD_SCALE_MASK_SOFTMAX_DROPOUT_CUDA_KERNEL(half, bool)

#undef REGISTER_FUSED_BIAS_ADD_SCALE_MASK_SOFTMAX_DROPOUT_CUDA_KERNEL

}  // namespace cuda
}  // namespace oneflow
