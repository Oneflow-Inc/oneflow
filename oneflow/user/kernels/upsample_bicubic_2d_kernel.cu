#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/common/nd_index_offset_helper.h"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/user/kernels/upsample_kernel.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T>
__device__ void upsample_increment_value_bounded_cuda(T* data, int64_t width, int64_t height,
                                                      int64_t element, int64_t x, int64_t y,
                                                      T value) {
  int64_t access_x = max(min(x, width - 1), static_cast<int64_t>(0));
  int64_t access_y = max(min(y, height - 1), static_cast<int64_t>(0));
  cuda::atomic::FastAdd(data, access_y * width + access_x, element, value);
}

template<typename T>
__global__ void UpsampleBicubic2dForward(const int64_t elem_cnt, const T* in_dptr,
                                         const int64_t nbatch, const int64_t channels,
                                         const int64_t in_height, const int64_t in_width,
                                         const int64_t out_height, const int64_t out_width,
                                         const float scale_height, const float scale_width,
                                         bool align_corners, T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(idx, elem_cnt) {
    const int output_x = idx % out_width;
    const int output_y = idx / out_width;

    const T* in = in_dptr;
    T* out = out_dptr;

    const T real_x = GetAreaPixel(scale_width, output_x, align_corners, /*cubic=*/true);
    int64_t input_x = floor(1.0 * real_x);
    const T t_x = real_x - input_x;

    const T real_y = GetAreaPixel(scale_height, output_y, align_corners, /*cubic=*/true);
    int64_t input_y = floor(1.0 * real_y);
    const T t_y = real_y - input_y;

    for (int64_t c = 0; c < channels * nbatch; c++) {
      T coefficients[4];

      // Interpolate 4 times in the x direction
      for (int64_t i = 0; i < 4; i++) {
        coefficients[i] = cubic_interp1d<T>(
            upsample_get_value_bounded<T>(in, in_width, in_height, input_x - 1, input_y - 1 + i),
            upsample_get_value_bounded<T>(in, in_width, in_height, input_x + 0, input_y - 1 + i),
            upsample_get_value_bounded<T>(in, in_width, in_height, input_x + 1, input_y - 1 + i),
            upsample_get_value_bounded<T>(in, in_width, in_height, input_x + 2, input_y - 1 + i),
            t_x);
      }

      // Interpolate in the y direction using x interpolations
      out[output_y * out_width + output_x] = cubic_interp1d<T>(
          coefficients[0], coefficients[1], coefficients[2], coefficients[3], t_y);

      // Move to next channel
      in += in_width * in_height;
      out += out_width * out_height;
    }
  }
}

template<typename T>
__global__ void UpsampleBicubic2dBackward(const int64_t elem_cnt, const T* dy_dptr,
                                          const int64_t nbatch, const int64_t channels,
                                          const int64_t in_height, const int64_t in_width,
                                          const int64_t out_height, const int64_t out_width,
                                          const float scale_height, const float scale_width,
                                          bool align_corners, T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(idx, elem_cnt) {
    const int output_x = idx % out_width;
    const int output_y = idx / out_width;

    T* in = dx_dptr;
    const T* out = dy_dptr;

    T real_x = GetAreaPixel(scale_width, output_x, align_corners, true);
    int64_t input_x = floor(1.0 * real_x);
    T t_x = real_x - input_x;

    T real_y = GetAreaPixel(scale_height, output_y, align_corners, true);
    int64_t input_y = floor(1.0 * real_y);
    T t_y = real_y - input_y;

    T x_coeffs[4];
    T y_coeffs[4];

    get_cubic_upsample_coefficients<T>(x_coeffs, t_x);
    get_cubic_upsample_coefficients<T>(y_coeffs, t_y);

    for (int64_t c = 0; c < channels * nbatch; c++) {
      T out_value = out[output_y * out_width + output_x];

      for (int64_t i = 0; i < 4; i++) {
        for (int64_t j = 0; j < 4; j++) {
          upsample_increment_value_bounded_cuda<T>(in, in_width, in_height, elem_cnt,
                                                   input_x - 1 + i, input_y - 1 + j,
                                                   out_value * y_coeffs[j] * x_coeffs[i]);
        }
      }

      in += in_width * in_height;
      out += out_width * out_height;
    }
  }
}

}  // namespace

template<typename T>
class UpsampleBicubic2dGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleBicubic2dGPUKernel() = default;
  ~UpsampleBicubic2dGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_tensor = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_tensor = ctx->Tensor4ArgNameAndIndex("y", 0);
    const T* in_ptr = x_tensor->dptr<T>();
    T* out_ptr = y_tensor->mut_dptr<T>();
    const bool align_corners = ctx->Attr<bool>("align_corners");

    const int nbatch = x_tensor->shape_view().At(0);
    const int channels = x_tensor->shape_view().At(1);
    const int64_t in_height = x_tensor->shape_view().At(2);
    const int64_t in_width = x_tensor->shape_view().At(3);
    const int64_t out_height = y_tensor->shape_view().At(2);
    const int64_t out_width = y_tensor->shape_view().At(3);
    const std::vector<int64_t> output_size = ctx->Attr<std::vector<int64_t>>("output_size");
    double height_scale = ctx->Attr<double>("height_scale");
    double width_scale = ctx->Attr<double>("width_scale");
    if (!output_size.empty()) {
      height_scale = static_cast<double>(out_height) / static_cast<double>(in_height);
      width_scale = static_cast<double>(out_width) / static_cast<double>(in_width);
    }
    const int64_t elem_cnt = out_height * out_width;

    if (in_height == out_height && in_width == out_width) {
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), y_tensor->mut_dptr<void>(), x_tensor->dptr<void>(),
          x_tensor->shape_view().elem_cnt() * GetSizeOfDataType(x_tensor->data_type()));
    } else {
      const T scale_height = GetAreaPixelScale(in_height, out_height, align_corners, height_scale);
      const T scale_width = GetAreaPixelScale(in_width, out_width, align_corners, width_scale);

      RUN_CUDA_KERNEL((UpsampleBicubic2dForward<T>), ctx->stream(), elem_cnt, elem_cnt,
                      x_tensor->dptr<T>(), nbatch, channels, in_height, in_width, out_height,
                      out_width, scale_height, scale_width, align_corners, y_tensor->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleBicubic2dGradGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleBicubic2dGradGPUKernel() = default;
  ~UpsampleBicubic2dGradGPUKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);
    Memset<DeviceType::kCUDA>(ctx->stream(), dx_tensor->mut_dptr<T>(), 0,
                              dx_tensor->shape_view().elem_cnt() * sizeof(T));
    const user_op::Tensor* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const bool align_corners = ctx->Attr<bool>("align_corners");

    const int nbatch = dx_tensor->shape_view().At(0);
    const int channels = dx_tensor->shape_view().At(1);
    const int64_t in_height = dx_tensor->shape_view().At(2);
    const int64_t in_width = dx_tensor->shape_view().At(3);
    const int64_t out_height = dy_tensor->shape_view().At(2);
    const int64_t out_width = dy_tensor->shape_view().At(3);
    const std::vector<int64_t> output_size = ctx->Attr<std::vector<int64_t>>("output_size");
    double height_scale = ctx->Attr<double>("height_scale");
    double width_scale = ctx->Attr<double>("width_scale");
    if (!output_size.empty()) {
      height_scale = static_cast<double>(out_height) / static_cast<double>(in_height);
      width_scale = static_cast<double>(out_width) / static_cast<double>(in_width);
    }
    const int64_t elem_cnt = out_height * out_width;

    if (in_height == out_height && in_width == out_width) {
      Memcpy<DeviceType::kCUDA>(
          ctx->stream(), dx_tensor->mut_dptr<void>(), dy_tensor->dptr<void>(),
          dy_tensor->shape_view().elem_cnt() * GetSizeOfDataType(dy_tensor->data_type()));
    } else {
      const T scale_height = GetAreaPixelScale(in_height, out_height, align_corners, height_scale);
      const T scale_width = GetAreaPixelScale(in_width, out_width, align_corners, width_scale);

      RUN_CUDA_KERNEL((UpsampleBicubic2dBackward<T>), ctx->stream(), elem_cnt, elem_cnt,
                      dy_tensor->dptr<T>(), nbatch, channels, in_height, in_width, out_height,
                      out_width, scale_height, scale_width, align_corners,
                      dx_tensor->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPLE_BICUBIC_CUDA_KERNEL(dtype)                                    \
  REGISTER_USER_KERNEL("upsample_bicubic_2d")                                           \
      .SetCreateFn<UpsampleBicubic2dGPUKernel<dtype>>()                                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)); \
  REGISTER_USER_KERNEL("upsample_bicubic_2d_grad")                                      \
      .SetCreateFn<UpsampleBicubic2dGradGPUKernel<dtype>>()                             \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_UPSAMPLE_BICUBIC_CUDA_KERNEL(float)
REGISTER_UPSAMPLE_BICUBIC_CUDA_KERNEL(double)

}  // namespace oneflow
