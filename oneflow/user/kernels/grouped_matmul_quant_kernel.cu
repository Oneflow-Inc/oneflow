
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#ifdef WITH_CUTLASS_EXTENSION

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/common/scalar.h"

#include <cutlass/library/library.h>
#include <cutlass/library/operation_table.h>
#include <cutlass/library/cutlass_extension_library.h>
#include "oneflow/user/kernels/cutlass_gemm_array_tuner.h"
#include <nlohmann/json.hpp>

namespace oneflow {

struct GemmProblem {
  GemmProblem(int64_t m, int64_t n, int64_t k) : m(m), n(n), k(k) {}
  int64_t m;
  int64_t n;
  int64_t k;
};

inline bool operator==(const GemmProblem& lhs, const GemmProblem& rhs) {
  return lhs.m == rhs.m && lhs.n == rhs.n && lhs.k == rhs.k;
}

}  // namespace oneflow

namespace std {

template<>
struct hash<oneflow::GemmProblem> {
  std::size_t operator()(const oneflow::GemmProblem& p) const {
    return oneflow::Hash<int64_t, int64_t, int64_t>(p.m, p.n, p.k);
  }
};

}  // namespace std

namespace oneflow {

namespace {

constexpr int64_t kMaxProblemBatch = 32;

template<typename T>
struct Buffer {
  const int8_t* a;
  const int8_t* b;
  const int8_t* in_zero_point;
  const float* in_scale;
  const T* weight_scale;
  const T* weight_acc;
  const T* scale;
  const T* biase;
  const T* _add_to_output;
  T* output;
};

template<typename T>
struct Param {
  Param(const GemmProblem& problem, std::vector<Buffer<T>> buffers)
      : problem(problem), batch_count(buffers.size()) {
    std::copy(buffers.cbegin(), buffers.cend(), buffer);
  }
  GemmProblem problem;
  Buffer<T> buffer[kMaxProblemBatch];
  int batch_count;
};

template<typename T>
__global__ void InitPtr(Param<T> p, void** ptr_arr) {
  CUDA_1D_KERNEL_LOOP(i, p.batch_count) {
    ptr_arr[i] = const_cast<int8_t*>(p.buffer[i].a);
    ptr_arr[i + kMaxProblemBatch] = const_cast<int8_t*>(p.buffer[i].b);
    ptr_arr[i + 2 * kMaxProblemBatch] = const_cast<int8_t*>(p.buffer[i].in_zero_point);
    ptr_arr[i + 3 * kMaxProblemBatch] = const_cast<float*>(p.buffer[i].in_scale);
    ptr_arr[i + 4 * kMaxProblemBatch] = const_cast<T*>(p.buffer[i].weight_scale);
    ptr_arr[i + 5 * kMaxProblemBatch] = const_cast<T*>(p.buffer[i].weight_acc);
    ptr_arr[i + 6 * kMaxProblemBatch] = const_cast<T*>(p.buffer[i].scale);
    ptr_arr[i + 7 * kMaxProblemBatch] = const_cast<T*>(p.buffer[i].biase);
    ptr_arr[i + 8 * kMaxProblemBatch] = const_cast<T*>(p.buffer[i]._add_to_output);
    ptr_arr[i + 9 * kMaxProblemBatch] = p.buffer[i].output;
  }
}

template<typename T>
void ApplyGroup(user_op::KernelComputeContext* ctx, const cutlass::library::GemmFunctionalKey& key,
                const GemmProblem& problem, bool has_in_zero_points, bool has_sacles,
                bool has_biases, bool has_add_to_outputs, std::vector<Buffer<T>> ptrs,
                user_op::Tensor* tmp_buffer, ep::Stream* stream) {
  void* tmp_ptr = tmp_buffer->mut_dptr();
  void** ptr_arr = reinterpret_cast<void**>(tmp_ptr);
  void* workspace = tmp_buffer + kMaxProblemBatch * 10 * sizeof(void*);
  size_t workspace_size =
      tmp_buffer->shape_view().elem_cnt() - kMaxProblemBatch * 10 * sizeof(void*);
  Param<T> params(problem, ptrs);
  RUN_CUDA_KERNEL((InitPtr<T>), stream, params.batch_count, params, ptr_arr);

  cutlass::gemm::GemmCoord problem_size(problem.m, problem.n, problem.k);

  cutlass::library::GemmArrayScaleBiasFusionConfiguration configuraion;
  configuraion.problem_size = problem_size;
  configuraion.lda = problem_size.k();
  configuraion.ldb = problem_size.k();
  configuraion.ld_filter_scale = 0;
  configuraion.ld_filter_acc = 0;
  configuraion.ld_scale = 0;
  configuraion.ld_bias = 0;
  configuraion.ldr = problem_size.n();
  configuraion.ldd = problem_size.n();
  configuraion.batch_count = params.batch_count;

  cutlass::library::GemmArrayScaleBiasFusionArguments arguments;
  arguments.A = ptr_arr;
  arguments.B = ptr_arr + kMaxProblemBatch;
  arguments.D = ptr_arr + 9 * kMaxProblemBatch;
  arguments.P = nullptr;
  arguments.InScale = nullptr;
  arguments.FilterScale = nullptr;
  arguments.FilterAcc = nullptr;
  arguments.Scale = nullptr;
  arguments.Bias = nullptr;
  arguments.Residual = nullptr;
  if (has_in_zero_points) {
    arguments.P = ptr_arr + 2 * kMaxProblemBatch;
    arguments.InScale = ptr_arr + 3 * kMaxProblemBatch;
    arguments.FilterScale = ptr_arr + 4 * kMaxProblemBatch;
    arguments.FilterAcc = ptr_arr + 5 * kMaxProblemBatch;
  }
  if (has_sacles) { arguments.Scale = ptr_arr + 6 * kMaxProblemBatch; }
  if (has_biases) { arguments.Bias = ptr_arr + 7 * kMaxProblemBatch; }
  if (has_add_to_outputs) { arguments.Residual = ptr_arr + 8 * kMaxProblemBatch; }

  auto* cuda_stream = stream->As<ep::CudaStream>();
  const cutlass::library::Operation* operation = nullptr;

  operation = [&]() -> const cutlass::library::Operation* {
    const std::string& tuning_cache = ctx->Attr<std::string>("tuning_cache");
    if (tuning_cache.empty()) { return nullptr; }
    auto tuning_cache_object = nlohmann::json::parse(tuning_cache);
    if (!tuning_cache_object.is_object()) { return nullptr; }
    auto it = tuning_cache_object.find("cutlass");
    if (it == tuning_cache_object.end()) { return nullptr; }
    if (!it->is_string()) { return nullptr; }
    const std::string name = *it;
    return CutlassGemmArrayTuner().GetOperation(name, cuda_stream, key, configuraion, arguments,
                                                workspace, workspace_size);
  }();
  if (!operation) {
    operation = CutlassGemmArrayTuner().FindOperation(cuda_stream, key, configuraion, arguments,
                                                      workspace, workspace_size);
  }
  CHECK(operation != nullptr);
  const size_t host_workspace_size = operation->get_host_workspace_size(&configuraion);
  std::vector<uint8_t> host_workspace(host_workspace_size, 0);

  auto init_status = operation->initialize(&configuraion, host_workspace.data(), workspace,
                                           cuda_stream->cuda_stream());
  CHECK(init_status == cutlass::Status::kSuccess);
  auto run_status =
      operation->run(&arguments, host_workspace.data(), workspace, cuda_stream->cuda_stream());
  CHECK(run_status == cutlass::Status::kSuccess);
}

template<typename OutType>
class GroupedMatmulQuantKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  GroupedMatmulQuantKernel() = default;
  ~GroupedMatmulQuantKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache* cache) const override {
    HashMap<GemmProblem, std::vector<Buffer<OutType>>> groups;
    const int32_t input_size = ctx->input_size("as");
    CHECK_EQ(ctx->input_size("bs"), input_size);
    const bool has_in_zero_points = ctx->has_input("in_zero_points", 0);
    const bool has_sacles = ctx->has_input("scales", 0);
    const bool has_biases = ctx->has_input("biases", 0);
    const bool has_add_to_outputs = ctx->has_input("_add_to_outputs", 0);

    cutlass::library::GemmFunctionalKey key(
        cutlass::library::Provider::kCUTLASS, cutlass::library::GemmKind::kGemm,
        cutlass::library::NumericTypeID::kS32,         // element_compute
        cutlass::library::NumericTypeID::kS32,         // element_scalar
        cutlass::library::NumericTypeID::kS8,          // element_A
        cutlass::library::LayoutTypeID::kRowMajor,     // layout_A
        cutlass::library::ComplexTransform::kNone,     // transform_A
        cutlass::library::NumericTypeID::kS8,          // element_B
        cutlass::library::LayoutTypeID::kColumnMajor,  // layout_B
        cutlass::library::ComplexTransform::kNone,     // transform_B
        cutlass::library::NumericTypeID::kS32,         // element_C
        cutlass::library::LayoutTypeID::kRowMajor,     // layout_C
        cutlass::library::NumericTypeID::kS32,         // element_D
        cutlass::library::LayoutTypeID::kRowMajor      // layout_D
    );

    if (GetDataType<OutType>::value == DataType::kFloat) {
      key.element_scalar = cutlass::library::NumericTypeID::kF32;
      key.element_C = cutlass::library::NumericTypeID::kF32;
      key.element_D = cutlass::library::NumericTypeID::kF32;
    } else if (GetDataType<OutType>::value == DataType::kFloat16) {
      key.element_scalar = cutlass::library::NumericTypeID::kF32;
      key.element_C = cutlass::library::NumericTypeID::kF16;
      key.element_D = cutlass::library::NumericTypeID::kF16;
    }

    for (int32_t i = 0; i < input_size; ++i) {
      const user_op::Tensor* a = ctx->Tensor4ArgNameAndIndex("as", i);
      const user_op::Tensor* b = ctx->Tensor4ArgNameAndIndex("bs", i);
      const user_op::Tensor* in_zero_point = ctx->Tensor4ArgNameAndIndex("in_zero_points", i);
      const user_op::Tensor* in_scale = ctx->Tensor4ArgNameAndIndex("in_scales", i);
      const user_op::Tensor* weight_scale = ctx->Tensor4ArgNameAndIndex("weight_scales", i);
      const user_op::Tensor* weight_acc = ctx->Tensor4ArgNameAndIndex("weight_accs", i);
      const user_op::Tensor* scale = ctx->Tensor4ArgNameAndIndex("scales", i);
      const user_op::Tensor* bias = ctx->Tensor4ArgNameAndIndex("biases", i);
      const user_op::Tensor* add_to_output = ctx->Tensor4ArgNameAndIndex("_add_to_outputs", i);
      user_op::Tensor* output = ctx->Tensor4ArgNameAndIndex("outputs", i);

      CHECK_GE(a->shape_view().NumAxes(), 2);
      const int64_t k = a->shape_view().At(a->shape_view().NumAxes() - 1);
      const int64_t m = a->shape_view().elem_cnt() / k;
      const int64_t n = b->shape_view().At(0);

      CHECK_EQ(output->shape_view().NumAxes(), a->shape_view().NumAxes());
      CHECK_EQ(output->shape_view().At(output->shape_view().NumAxes() - 1), n);
      for (int32_t j = 0; j < output->shape_view().NumAxes() - 1; ++j) {
        CHECK_EQ(output->shape_view().At(j), a->shape_view().At(j));
      }
      const int8_t* a_ptr = a->dptr<int8_t>();
      const int8_t* b_ptr = b->dptr<int8_t>();
      const int8_t* in_zero_point_ptr =
          has_in_zero_points ? in_zero_point->dptr<int8_t>() : nullptr;
      const float* in_scale_ptr = has_in_zero_points ? in_scale->dptr<float>() : nullptr;
      const OutType* weight_scale_ptr =
          has_in_zero_points ? weight_scale->dptr<OutType>() : nullptr;
      const OutType* weight_acc_ptr = has_in_zero_points ? weight_acc->dptr<OutType>() : nullptr;
      const OutType* scale_ptr = has_sacles ? scale->dptr<OutType>() : nullptr;
      const OutType* bias_ptr = has_biases ? bias->dptr<OutType>() : nullptr;
      const OutType* add_to_output_ptr =
          has_add_to_outputs ? add_to_output->dptr<OutType>() : nullptr;
      OutType* output_ptr = output->mut_dptr<OutType>();

      groups[GemmProblem(m, n, k)].push_back(
          Buffer<OutType>{a_ptr, b_ptr, in_zero_point_ptr, in_scale_ptr, weight_scale_ptr,
                          weight_acc_ptr, scale_ptr, bias_ptr, add_to_output_ptr, output_ptr});
    }
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    for (const auto& group : groups) {
      for (size_t i = 0; i < group.second.size(); i += kMaxProblemBatch) {
        std::vector<Buffer<OutType>> ptrs(
            {group.second.begin() + i,
             group.second.begin() + i
                 + std::min<size_t>(group.second.size() - i, kMaxProblemBatch)});
        ApplyGroup<OutType>(ctx, key, group.first, has_in_zero_points, has_sacles, has_biases,
                            has_add_to_outputs, ptrs, tmp_buffer, ctx->stream());
      }
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GROUPED_MATMUL_BIAS_KERNEL_GPU(out_cpp_type, out_data_type)     \
  REGISTER_USER_KERNEL("grouped_matmul_quant")                                   \
      .SetCreateFn<GroupedMatmulQuantKernel<out_cpp_type>>()                     \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)           \
                       && (user_op::HobDataType("as", 0) == DataType::kInt8)     \
                       && (user_op::HobDataType("bs", 0) == DataType::kInt8)     \
                       && (user_op::HobDataType("outputs", 0) == out_data_type)) \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) -> size_t {              \
        return kMaxProblemBatch * 10 * sizeof(void*) + 3 * 1024 * 1024;          \
      });

REGISTER_GROUPED_MATMUL_BIAS_KERNEL_GPU(half, DataType::kFloat16)
REGISTER_GROUPED_MATMUL_BIAS_KERNEL_GPU(float, DataType::kFloat)

}  // namespace

}  // namespace oneflow

#endif  // WITH_CUTLASS_EXTENSION
