#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/common/nd_index_offset_helper.h"
#include "oneflow/core/cuda/atomic.cuh"

namespace oneflow {

namespace {

__device__ int64_t GetNearestInputIndex(const int64_t out_dim_idx, const float scale,
                                        const int64_t in_dim_size) {
  return max(min(static_cast<int64_t>(floorf((static_cast<float>(out_dim_idx) + 0.5f) * scale)),
                 in_dim_size - 1),
             static_cast<int64_t>(0));
}

template<typename T>
__global__ void UpsampleNearestForward(const int64_t elem_cnt, const T* in_dptr,
                                       NdIndexOffsetHelper<int64_t, 4> in_helper,
                                       NdIndexOffsetHelper<int64_t, 4> out_helper,
                                       const int64_t in_height, const int64_t in_width,
                                       const float scale_h, const float scale_w, T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    out_helper.OffsetToNdIndex(index, n, c, h, w);
    const int64_t in_h = GetNearestInputIndex(h, scale_h, in_height);
    const int64_t in_w = GetNearestInputIndex(w, scale_w, in_width);
    out_dptr[index] = in_dptr[in_helper.NdIndexToOffset(n, c, in_h, in_w)];
  }
}

template<typename T>
__global__ void UpsampleNearestBackward(const int64_t elem_cnt, const T* dy_dptr,
                                        NdIndexOffsetHelper<int64_t, 4> dy_helper,
                                        NdIndexOffsetHelper<int64_t, 4> dx_helper,
                                        const int64_t dx_height, const int64_t dx_width,
                                        const float scale_h, const float scale_w, T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    dy_helper.OffsetToNdIndex(index, n, c, h, w);
    const int64_t dx_h = GetNearestInputIndex(h, scale_h, dx_height);
    const int64_t dx_w = GetNearestInputIndex(w, scale_w, dx_width);
    cuda::atomic::Add(dx_dptr + dx_helper.NdIndexToOffset(n, c, dx_h, dx_w), dy_dptr[index]);
  }
}

struct BilinearParam {
  int64_t top_h_index;
  int64_t bottom_h_index;
  int64_t left_w_index;
  int64_t right_w_index;
  float w_lerp;
  float h_lerp;
};

__device__ void GetBilinearParam(const int64_t index, const int64_t h, const int64_t w,
                                 const int64_t in_height, const int64_t in_width,
                                 const float scale_h, const float scale_w, BilinearParam* params) {
  const float in_h = (static_cast<float>(h) + 0.5f) * scale_h - 0.5f;
  const float in_w = (static_cast<float>(w) + 0.5f) * scale_w - 0.5f;
  params->top_h_index = in_h > 0.0 ? floorf(in_h) : 0;
  params->bottom_h_index = (in_h < in_height - 1) ? ceilf(in_h) : in_height - 1;
  params->h_lerp = in_h - floorf(in_h);
  params->left_w_index = in_w > 0.0 ? floorf(in_w) : 0;
  params->right_w_index = (in_w < in_width - 1) ? ceilf(in_w) : in_width - 1;
  params->w_lerp = in_w - floorf(in_w);
}

template<typename T>
__global__ void UpsampleBilinearForward(const int64_t elem_cnt, const T* in_dptr,
                                        NdIndexOffsetHelper<int64_t, 4> in_helper,
                                        NdIndexOffsetHelper<int64_t, 4> out_helper,
                                        const int64_t in_height, const int64_t in_width,
                                        const float scale_h, const float scale_w, T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    out_helper.OffsetToNdIndex(index, n, c, h, w);
    BilinearParam params;
    GetBilinearParam(index, h, w, in_height, in_width, scale_h, scale_w, &params);
    const int64_t top_offset = in_helper.NdIndexToOffset(n, c, params.top_h_index, 0);
    const int64_t bottom_offset = in_helper.NdIndexToOffset(n, c, params.bottom_h_index, 0);
    const float top_left = in_dptr[top_offset + params.left_w_index];
    const float top_right = in_dptr[top_offset + params.right_w_index];
    const float bottom_left = in_dptr[bottom_offset + params.left_w_index];
    const float bottom_right = in_dptr[bottom_offset + params.right_w_index];
    const float top = top_left + (top_right - top_left) * params.w_lerp;
    const float bottom = bottom_left + (bottom_right - bottom_left) * params.w_lerp;
    out_dptr[index] = top + (bottom - top) * params.h_lerp;
  }
}

template<typename T>
__global__ void UpsampleBilinearBackward(const int64_t elem_cnt, const T* dy_dptr,
                                         NdIndexOffsetHelper<int64_t, 4> dy_helper,
                                         NdIndexOffsetHelper<int64_t, 4> dx_helper,
                                         const int64_t dx_height, const int64_t dx_width,
                                         const float scale_h, const float scale_w, T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    dy_helper.OffsetToNdIndex(index, n, c, h, w);
    BilinearParam params;
    GetBilinearParam(index, h, w, dx_height, dx_width, scale_h, scale_w, &params);
    const int64_t top_offset = dx_helper.NdIndexToOffset(n, c, params.top_h_index, 0);
    const int64_t bottom_offset = dx_helper.NdIndexToOffset(n, c, params.bottom_h_index, 0);
    const T dy = dy_dptr[index];
    const float dbottom = params.h_lerp * dy;
    T* dx_dptr_bottom_offset = dx_dptr + bottom_offset;
    cuda::atomic::Add(dx_dptr_bottom_offset + params.left_w_index,
                      static_cast<T>((1 - params.w_lerp) * dbottom));
    cuda::atomic::Add(dx_dptr_bottom_offset + params.right_w_index,
                      static_cast<T>(params.w_lerp * dbottom));
    const float dtop = dy - dbottom;
    T* dx_dptr_top_offset = dx_dptr + top_offset;
    cuda::atomic::Add(dx_dptr_top_offset + params.left_w_index,
                      static_cast<T>((1 - params.w_lerp) * dtop));
    cuda::atomic::Add(dx_dptr_top_offset + params.right_w_index,
                      static_cast<T>(params.w_lerp * dtop));
  }
}

}  // namespace

template<typename T>
class UpsampleNearestGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearestGPUKernel() = default;
  ~UpsampleNearestGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_blob = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_blob = ctx->Tensor4ArgNameAndIndex("y", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const int64_t elem_cnt = y_blob->shape().elem_cnt();
    NdIndexOffsetHelper<int64_t, 4> in_helper(x_blob->shape().At(0), x_blob->shape().At(1),
                                              x_blob->shape().At(2), x_blob->shape().At(3));
    NdIndexOffsetHelper<int64_t, 4> out_helper(y_blob->shape().At(0), y_blob->shape().At(1),
                                               y_blob->shape().At(2), y_blob->shape().At(3));

    RUN_CUDA_KERNEL((UpsampleNearestForward<T>), ctx->device_ctx(), elem_cnt, elem_cnt,
                    x_blob->dptr<T>(), in_helper, out_helper, x_blob->shape().At(2),
                    x_blob->shape().At(3), 1.f / height_scale, 1.f / width_scale,
                    y_blob->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleNearestGradGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearestGradGPUKernel() = default;
  ~UpsampleNearestGradGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_blob = ctx->Tensor4ArgNameAndIndex("dx", 0);
    if (dx_blob == nullptr) { return; }
    Memset<DeviceType::kGPU>(ctx->device_ctx(), dx_blob->mut_dptr<T>(), 0,
                             dx_blob->shape().elem_cnt() * sizeof(T));
    const user_op::Tensor* dy_blob = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const int64_t elem_cnt = dy_blob->shape().elem_cnt();
    NdIndexOffsetHelper<int64_t, 4> dy_helper(dy_blob->shape().At(0), dy_blob->shape().At(1),
                                              dy_blob->shape().At(2), dy_blob->shape().At(3));
    NdIndexOffsetHelper<int64_t, 4> dx_helper(dx_blob->shape().At(0), dx_blob->shape().At(1),
                                              dx_blob->shape().At(2), dx_blob->shape().At(3));
    RUN_CUDA_KERNEL((UpsampleNearestBackward<T>), ctx->device_ctx(), elem_cnt, elem_cnt,
                    dy_blob->dptr<T>(), dy_helper, dx_helper, dx_blob->shape().At(2),
                    dx_blob->shape().At(3), 1.f / height_scale, 1.f / width_scale,
                    dx_blob->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPLE_NEAREST_GPU_KERNEL(dtype)                                      \
  REGISTER_USER_KERNEL("upsample")                                                       \
      .SetCreateFn<UpsampleNearestGPUKernel<dtype>>()                                    \
      .SetIsMatchedHob(                                                                  \
          (user_op::HobDeviceTag() == "gpu")                                             \
          & (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)                  \
          & (user_op::HobAttr<std::string>("interpolation") == std::string("nearest"))); \
  REGISTER_USER_KERNEL("upsample_grad")                                                  \
      .SetCreateFn<UpsampleNearestGradGPUKernel<dtype>>()                                \
      .SetIsMatchedHob(                                                                  \
          (user_op::HobDeviceTag() == "gpu")                                             \
          & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value)                 \
          & (user_op::HobAttr<std::string>("interpolation") == std::string("nearest")));

REGISTER_UPSAMPLE_NEAREST_GPU_KERNEL(float)
REGISTER_UPSAMPLE_NEAREST_GPU_KERNEL(double)

template<typename T>
class UpsampleBilinearGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleBilinearGPUKernel() = default;
  ~UpsampleBilinearGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_blob = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_blob = ctx->Tensor4ArgNameAndIndex("y", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const int64_t elem_cnt = y_blob->shape().elem_cnt();
    NdIndexOffsetHelper<int64_t, 4> in_helper(x_blob->shape().At(0), x_blob->shape().At(1),
                                              x_blob->shape().At(2), x_blob->shape().At(3));
    NdIndexOffsetHelper<int64_t, 4> out_helper(y_blob->shape().At(0), y_blob->shape().At(1),
                                               y_blob->shape().At(2), y_blob->shape().At(3));

    RUN_CUDA_KERNEL((UpsampleBilinearForward<T>), ctx->device_ctx(), elem_cnt, elem_cnt,
                    x_blob->dptr<T>(), in_helper, out_helper, x_blob->shape().At(2),
                    x_blob->shape().At(3), 1.f / height_scale, 1.f / width_scale,
                    y_blob->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleBilinearGradGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleBilinearGradGPUKernel() = default;
  ~UpsampleBilinearGradGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_blob = ctx->Tensor4ArgNameAndIndex("dx", 0);
    if (dx_blob == nullptr) { return; }
    Memset<DeviceType::kGPU>(ctx->device_ctx(), dx_blob->mut_dptr<T>(), 0,
                             dx_blob->shape().elem_cnt() * sizeof(T));
    const user_op::Tensor* dy_blob = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const int64_t elem_cnt = dy_blob->shape().elem_cnt();
    NdIndexOffsetHelper<int64_t, 4> dy_helper(dy_blob->shape().At(0), dy_blob->shape().At(1),
                                              dy_blob->shape().At(2), dy_blob->shape().At(3));
    NdIndexOffsetHelper<int64_t, 4> dx_helper(dx_blob->shape().At(0), dx_blob->shape().At(1),
                                              dx_blob->shape().At(2), dx_blob->shape().At(3));

    RUN_CUDA_KERNEL((UpsampleBilinearBackward<T>), ctx->device_ctx(), elem_cnt, elem_cnt,
                    dy_blob->dptr<T>(), dy_helper, dx_helper, dx_blob->shape().At(2),
                    dx_blob->shape().At(3), 1.f / height_scale, 1.f / width_scale,
                    dx_blob->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPLE_BILINEAR_GPU_KERNEL(dtype)                                      \
  REGISTER_USER_KERNEL("upsample")                                                        \
      .SetCreateFn<UpsampleBilinearGPUKernel<dtype>>()                                    \
      .SetIsMatchedHob(                                                                   \
          (user_op::HobDeviceTag() == "gpu")                                              \
          & (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)                   \
          & (user_op::HobAttr<std::string>("interpolation") == std::string("bilinear"))); \
  REGISTER_USER_KERNEL("upsample_grad")                                                   \
      .SetCreateFn<UpsampleBilinearGradGPUKernel<dtype>>()                                \
      .SetIsMatchedHob(                                                                   \
          (user_op::HobDeviceTag() == "gpu")                                              \
          & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value)                  \
          & (user_op::HobAttr<std::string>("interpolation") == std::string("bilinear")));

REGISTER_UPSAMPLE_BILINEAR_GPU_KERNEL(float)
REGISTER_UPSAMPLE_BILINEAR_GPU_KERNEL(double)

}  // namespace oneflow
