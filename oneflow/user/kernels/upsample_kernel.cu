#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/common/nd_index_offset_helper.h"
#include "oneflow/core/cuda/atomic.cuh"

namespace oneflow {

namespace {

__device__ int64_t GetNearestInputIndex(const int64_t out_dim_idx, const float scale,
                                        const int64_t in_dim_size) {
  return max(min(static_cast<int64_t>(floorf((static_cast<float>(out_dim_idx) + 0.5f) * scale)),
                 in_dim_size - 1),
             static_cast<int64_t>(0));
}

template<typename T>
__global__ void UpsampleNearestForward(const int64_t elem_cnt, const T* in_dptr,
                                       NdIndexOffsetHelper<int64_t, 4> in_helper,
                                       NdIndexOffsetHelper<int64_t, 4> out_helper,
                                       const int64_t in_height, const int64_t in_width,
                                       const float scale_h, const float scale_w, T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    out_helper.OffsetToNdIndex(index, n, c, h, w);
    const int64_t in_h = GetNearestInputIndex(h, scale_h, in_height);
    const int64_t in_w = GetNearestInputIndex(w, scale_w, in_width);
    out_dptr[index] = in_dptr[in_helper.NdIndexToOffset(n, c, in_h, in_w)];
  }
}

template<typename T>
__global__ void UpsampleNearestBackward(const int64_t elem_cnt, const T* dy_dptr,
                                        NdIndexOffsetHelper<int64_t, 4> dy_helper,
                                        NdIndexOffsetHelper<int64_t, 4> dx_helper,
                                        const int64_t dx_height, const int64_t dx_width,
                                        const float scale_h, const float scale_w, T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    dy_helper.OffsetToNdIndex(index, n, c, h, w);
    const int64_t dx_h = GetNearestInputIndex(h, scale_h, dx_height);
    const int64_t dx_w = GetNearestInputIndex(w, scale_w, dx_width);
    cuda::atomic::Add(dx_dptr + dx_helper.NdIndexToOffset(n, c, dx_h, dx_w), dy_dptr[index]);
  }
}

template<typename T>
__host__ T GetAreaPixelScale(const int64_t input_size, const int64_t output_size,
                             bool align_corners, const T scale) {
  return align_corners ? static_cast<T>(input_size - 1) / (output_size - 1)
                       : (scale > 0. ? 1.0 / scale : static_cast<T>(input_size) / output_size);
}

template<typename T>
__device__ T GetAreaPixelSourceIndex(const T scale, const int64_t dst_index, bool align_corners) {
  if (align_corners) {
    return scale * static_cast<T>(dst_index);
  } else {
    T src_index = (static_cast<T>(dst_index) + 0.5f) * scale - 0.5f;
    return (src_index < 0) ? 0 : src_index;
  }
}

template<typename T>
struct BilinearParam {
  int64_t top_h_index;
  int64_t bottom_h_index;
  int64_t left_w_index;
  int64_t right_w_index;
  T w_lerp;
  T h_lerp;
};

template<typename T>
__device__ void GetBilinearParam(const bool align_corners, const int64_t h, const int64_t w,
                                 const int64_t in_height, const int64_t in_width, const T scale_h,
                                 const T scale_w, BilinearParam<T>* params) {
  const T in_h = GetAreaPixelSourceIndex(scale_h, h, align_corners);
  const T in_w = GetAreaPixelSourceIndex(scale_w, w, align_corners);
  params->top_h_index = in_h > 0.0 ? floorf(in_h) : 0;
  params->bottom_h_index = (in_h < in_height - 1) ? ceilf(in_h) : in_height - 1;
  params->h_lerp = in_h - floorf(in_h);
  params->left_w_index = in_w > 0.0 ? floorf(in_w) : 0;
  params->right_w_index = (in_w < in_width - 1) ? ceilf(in_w) : in_width - 1;
  params->w_lerp = in_w - floorf(in_w);
}

template<typename T>
__global__ void UpsampleBilinearForward(const int64_t elem_cnt, const T* in_dptr,
                                        NdIndexOffsetHelper<int64_t, 4> in_helper,
                                        NdIndexOffsetHelper<int64_t, 4> out_helper,
                                        const int64_t in_height, const int64_t in_width,
                                        const T scale_h, const T scale_w, const bool align_corners,
                                        T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    out_helper.OffsetToNdIndex(index, n, c, h, w);
    BilinearParam<T> params;
    GetBilinearParam(align_corners, h, w, in_height, in_width, scale_h, scale_w, &params);
    const int64_t top_offset = in_helper.NdIndexToOffset(n, c, params.top_h_index, 0);
    const int64_t bottom_offset = in_helper.NdIndexToOffset(n, c, params.bottom_h_index, 0);
    const T top_left = in_dptr[top_offset + params.left_w_index];
    const T top_right = in_dptr[top_offset + params.right_w_index];
    const T bottom_left = in_dptr[bottom_offset + params.left_w_index];
    const T bottom_right = in_dptr[bottom_offset + params.right_w_index];
    const T top = top_left + (top_right - top_left) * params.w_lerp;
    const T bottom = bottom_left + (bottom_right - bottom_left) * params.w_lerp;
    out_dptr[index] = top + (bottom - top) * params.h_lerp;
  }
}

template<typename T>
__global__ void UpsampleBilinearBackward(const int64_t elem_cnt, const T* dy_dptr,
                                         NdIndexOffsetHelper<int64_t, 4> dy_helper,
                                         NdIndexOffsetHelper<int64_t, 4> dx_helper,
                                         const int64_t dx_height, const int64_t dx_width,
                                         const T scale_h, const T scale_w, const bool align_corners,
                                         T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h, w;
    dy_helper.OffsetToNdIndex(index, n, c, h, w);
    BilinearParam<T> params;
    GetBilinearParam(align_corners, h, w, dx_height, dx_width, scale_h, scale_w, &params);
    const int64_t top_offset = dx_helper.NdIndexToOffset(n, c, params.top_h_index, 0);
    const int64_t bottom_offset = dx_helper.NdIndexToOffset(n, c, params.bottom_h_index, 0);
    const T dy = dy_dptr[index];
    const T dbottom = params.h_lerp * dy;
    T* dx_dptr_bottom_offset = dx_dptr + bottom_offset;
    cuda::atomic::Add(dx_dptr_bottom_offset + params.left_w_index,
                      static_cast<T>((1 - params.w_lerp) * dbottom));
    cuda::atomic::Add(dx_dptr_bottom_offset + params.right_w_index,
                      static_cast<T>(params.w_lerp * dbottom));
    const T dtop = dy - dbottom;
    T* dx_dptr_top_offset = dx_dptr + top_offset;
    cuda::atomic::Add(dx_dptr_top_offset + params.left_w_index,
                      static_cast<T>((1 - params.w_lerp) * dtop));
    cuda::atomic::Add(dx_dptr_top_offset + params.right_w_index,
                      static_cast<T>(params.w_lerp * dtop));
  }
}

}  // namespace

template<typename T>
class UpsampleNearestGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearestGPUKernel() = default;
  ~UpsampleNearestGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_blob = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_blob = ctx->Tensor4ArgNameAndIndex("y", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const int64_t elem_cnt = y_blob->shape().elem_cnt();
    NdIndexOffsetHelper<int64_t, 4> in_helper(x_blob->shape().At(0), x_blob->shape().At(1),
                                              x_blob->shape().At(2), x_blob->shape().At(3));
    NdIndexOffsetHelper<int64_t, 4> out_helper(y_blob->shape().At(0), y_blob->shape().At(1),
                                               y_blob->shape().At(2), y_blob->shape().At(3));

    RUN_CUDA_KERNEL((UpsampleNearestForward<T>), ctx->device_ctx(), elem_cnt, elem_cnt,
                    x_blob->dptr<T>(), in_helper, out_helper, x_blob->shape().At(2),
                    x_blob->shape().At(3), 1.f / height_scale, 1.f / width_scale,
                    y_blob->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleNearestGradGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearestGradGPUKernel() = default;
  ~UpsampleNearestGradGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_blob = ctx->Tensor4ArgNameAndIndex("dx", 0);
    if (dx_blob == nullptr) { return; }
    Memset<DeviceType::kGPU>(ctx->device_ctx(), dx_blob->mut_dptr<T>(), 0,
                             dx_blob->shape().elem_cnt() * sizeof(T));
    const user_op::Tensor* dy_blob = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const int64_t elem_cnt = dy_blob->shape().elem_cnt();
    NdIndexOffsetHelper<int64_t, 4> dy_helper(dy_blob->shape().At(0), dy_blob->shape().At(1),
                                              dy_blob->shape().At(2), dy_blob->shape().At(3));
    NdIndexOffsetHelper<int64_t, 4> dx_helper(dx_blob->shape().At(0), dx_blob->shape().At(1),
                                              dx_blob->shape().At(2), dx_blob->shape().At(3));
    RUN_CUDA_KERNEL((UpsampleNearestBackward<T>), ctx->device_ctx(), elem_cnt, elem_cnt,
                    dy_blob->dptr<T>(), dy_helper, dx_helper, dx_blob->shape().At(2),
                    dx_blob->shape().At(3), 1.f / height_scale, 1.f / width_scale,
                    dx_blob->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPLE_NEAREST_GPU_KERNEL(dtype)                                      \
  REGISTER_USER_KERNEL("upsample")                                                       \
      .SetCreateFn<UpsampleNearestGPUKernel<dtype>>()                                    \
      .SetIsMatchedHob(                                                                  \
          (user_op::HobDeviceTag() == "gpu")                                             \
          & (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)                  \
          & (user_op::HobAttr<std::string>("interpolation") == std::string("nearest"))); \
  REGISTER_USER_KERNEL("upsample_grad")                                                  \
      .SetCreateFn<UpsampleNearestGradGPUKernel<dtype>>()                                \
      .SetIsMatchedHob(                                                                  \
          (user_op::HobDeviceTag() == "gpu")                                             \
          & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value)                 \
          & (user_op::HobAttr<std::string>("interpolation") == std::string("nearest")));

REGISTER_UPSAMPLE_NEAREST_GPU_KERNEL(float)
REGISTER_UPSAMPLE_NEAREST_GPU_KERNEL(double)

template<typename T>
class UpsampleBilinearGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleBilinearGPUKernel() = default;
  ~UpsampleBilinearGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_blob = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_blob = ctx->Tensor4ArgNameAndIndex("y", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const bool align_corners = ctx->Attr<bool>("align_corners");
    const int64_t elem_cnt = y_blob->shape().elem_cnt();
    NdIndexOffsetHelper<int64_t, 4> in_helper(x_blob->shape().At(0), x_blob->shape().At(1),
                                              x_blob->shape().At(2), x_blob->shape().At(3));
    NdIndexOffsetHelper<int64_t, 4> out_helper(y_blob->shape().At(0), y_blob->shape().At(1),
                                               y_blob->shape().At(2), y_blob->shape().At(3));

    const int64_t in_height = x_blob->shape().At(2);
    const int64_t in_width = x_blob->shape().At(3);
    const int64_t out_height = y_blob->shape().At(2);
    const int64_t out_width = y_blob->shape().At(3);
    const T scale_height = GetAreaPixelScale(in_height, out_height, align_corners, height_scale);
    const T scale_width = GetAreaPixelScale(in_width, out_width, align_corners, width_scale);
    RUN_CUDA_KERNEL((UpsampleBilinearForward<T>), ctx->device_ctx(), elem_cnt, elem_cnt,
                    x_blob->dptr<T>(), in_helper, out_helper, in_height, in_width, scale_height,
                    scale_width, align_corners, y_blob->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleBilinearGradGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleBilinearGradGPUKernel() = default;
  ~UpsampleBilinearGradGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_blob = ctx->Tensor4ArgNameAndIndex("dx", 0);
    if (dx_blob == nullptr) { return; }
    Memset<DeviceType::kGPU>(ctx->device_ctx(), dx_blob->mut_dptr<T>(), 0,
                             dx_blob->shape().elem_cnt() * sizeof(T));
    const user_op::Tensor* dy_blob = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const float height_scale = ctx->Attr<float>("height_scale");
    const float width_scale = ctx->Attr<float>("width_scale");
    const bool align_corners = ctx->Attr<bool>("align_corners");
    const int64_t elem_cnt = dy_blob->shape().elem_cnt();
    NdIndexOffsetHelper<int64_t, 4> dy_helper(dy_blob->shape().At(0), dy_blob->shape().At(1),
                                              dy_blob->shape().At(2), dy_blob->shape().At(3));
    NdIndexOffsetHelper<int64_t, 4> dx_helper(dx_blob->shape().At(0), dx_blob->shape().At(1),
                                              dx_blob->shape().At(2), dx_blob->shape().At(3));

    const int64_t in_height = dx_blob->shape().At(2);
    const int64_t in_width = dx_blob->shape().At(3);
    const int64_t out_height = dy_blob->shape().At(2);
    const int64_t out_width = dy_blob->shape().At(3);
    const T scale_height = GetAreaPixelScale(in_height, out_height, align_corners, height_scale);
    const T scale_width = GetAreaPixelScale(in_width, out_width, align_corners, width_scale);
    RUN_CUDA_KERNEL((UpsampleBilinearBackward<T>), ctx->device_ctx(), elem_cnt, elem_cnt,
                    dy_blob->dptr<T>(), dy_helper, dx_helper, in_height, in_width, scale_height,
                    scale_width, align_corners, dx_blob->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPLE_BILINEAR_GPU_KERNEL(dtype)                                      \
  REGISTER_USER_KERNEL("upsample")                                                        \
      .SetCreateFn<UpsampleBilinearGPUKernel<dtype>>()                                    \
      .SetIsMatchedHob(                                                                   \
          (user_op::HobDeviceTag() == "gpu")                                              \
          & (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)                   \
          & (user_op::HobAttr<std::string>("interpolation") == std::string("bilinear"))); \
  REGISTER_USER_KERNEL("upsample_grad")                                                   \
      .SetCreateFn<UpsampleBilinearGradGPUKernel<dtype>>()                                \
      .SetIsMatchedHob(                                                                   \
          (user_op::HobDeviceTag() == "gpu")                                              \
          & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value)                  \
          & (user_op::HobAttr<std::string>("interpolation") == std::string("bilinear")));

REGISTER_UPSAMPLE_BILINEAR_GPU_KERNEL(float)
REGISTER_UPSAMPLE_BILINEAR_GPU_KERNEL(double)

}  // namespace oneflow
