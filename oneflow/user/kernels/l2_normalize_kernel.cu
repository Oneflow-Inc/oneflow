#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/cuda/layer_norm.cuh"

namespace oneflow {

namespace {

template<typename T, typename ComputeType>
__global__ void L2NormalizeForward(const int32_t n, const int32_t c, const int32_t d,
                                   const ComputeType epsilon, const T* in,
                                   ComputeType* square_x_sum, T* out) {
  using BlockReduce = hipcub::BlockReduce<ComputeType, ep::CudaStream::kDefaultBlockSize>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  for (int32_t i = blockIdx.x; i < n; i += gridDim.x) {
    ComputeType sum = GetZeroVal<ComputeType>();
    const int32_t offset = (i / d) * d * c + (i % d);
    for (int32_t j = threadIdx.x; j < c; j += blockDim.x) {
      const ComputeType x = static_cast<ComputeType>(in[offset + j * d]);
      sum += x * x;
    }
    const ComputeType reduce_sum = BlockReduce(temp_storage).Sum(sum);
    if (threadIdx.x == 0) { square_x_sum[i] = reduce_sum; }
    __syncthreads();

    const ComputeType inv_norm = rsqrtf(fmaxf(square_x_sum[i], epsilon));
    for (int32_t j = threadIdx.x; j < c; j += blockDim.x) {
      const int32_t index = offset + j * d;
      out[index] = static_cast<T>(inv_norm * static_cast<ComputeType>(in[index]));
    }
  }
}

template<typename T>
__global__ void L2NormalizeBackward(const int32_t n, const int32_t c, const int32_t d,
                                    const float epsilon, const T* out, const T* out_diff,
                                    const T* square_x_sum, T* in_diff) {
  for (int32_t i = blockIdx.x; i < n; i += gridDim.x) {
    const T inv_norm = rsqrt(fmaxf(square_x_sum[i], epsilon));
    const int32_t offset = (i / d) * d * c + (i % d);
    if (square_x_sum[i] >= epsilon) {
      using BlockReduce = hipcub::BlockReduce<T, ep::CudaStream::kDefaultBlockSize>;
      __shared__ typename BlockReduce::TempStorage temp_storage_prod_sum;

      T y_dy_prod_sum = GetZeroVal<T>();
      for (int32_t j = threadIdx.x; j < c; j += blockDim.x) {
        const int32_t index = offset + j * d;
        y_dy_prod_sum += out[index] * out_diff[index];
      }

      const T reduce_y_dy_prod_sum = BlockReduce(temp_storage_prod_sum).Sum(y_dy_prod_sum);
      __shared__ T y_dy_inner_prod;
      if (threadIdx.x == 0) { y_dy_inner_prod = reduce_y_dy_prod_sum; }
      __syncthreads();

      for (int32_t j = threadIdx.x; j < c; j += blockDim.x) {
        const int32_t index = offset + j * d;
        in_diff[index] = inv_norm * (out_diff[index] - y_dy_inner_prod * out[index]);
      }
    } else {
      for (int32_t j = threadIdx.x; j < c; j += blockDim.x) {
        const int32_t index = offset + j * d;
        in_diff[index] = inv_norm * out_diff[index];
      }
    }
  }
}

}  // namespace

template<typename T>
class GpuL2NormalizeKernel final : public user_op::OpKernel {
 public:
  GpuL2NormalizeKernel() = default;
  ~GpuL2NormalizeKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    user_op::Tensor* square_x_sum = ctx->Tensor4ArgNameAndIndex("square_x_sum", 0);
    const float epsilon = ctx->Attr<float>("epsilon");
    int32_t axis = ctx->Attr<int32_t>("axis");
    int32_t c = x->shape_view().At(axis);
    int32_t n = x->shape_view().elem_cnt() / c;
    int32_t d = x->shape_view().Count(axis + 1);
    using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
    RUN_CUDA_KERNEL((L2NormalizeForward<T, ComputeType>), ctx->stream(), n, n, c, d,
                    static_cast<ComputeType>(epsilon), x->dptr<T>(),
                    square_x_sum->mut_dptr<ComputeType>(), y->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_L2_NORMALIZE_KERNEL(dtype)                       \
  REGISTER_USER_KERNEL("l2_normalize")                                 \
      .SetCreateFn<GpuL2NormalizeKernel<dtype>>()                      \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("y", 0) == GetDataType<dtype>::value));

REGISTER_CUDA_L2_NORMALIZE_KERNEL(half)
REGISTER_CUDA_L2_NORMALIZE_KERNEL(float)
REGISTER_CUDA_L2_NORMALIZE_KERNEL(double)

template<typename T>
class GpuL2NormalizeGradKernel final : public user_op::OpKernel {
 public:
  GpuL2NormalizeGradKernel() = default;
  ~GpuL2NormalizeGradKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* square_x_sum = ctx->Tensor4ArgNameAndIndex("square_x_sum", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const float epsilon = ctx->Attr<float>("epsilon");
    int32_t axis = ctx->Attr<int32_t>("axis");
    int32_t c = dy->shape_view().At(axis);
    int32_t n = dy->shape_view().elem_cnt() / c;
    int32_t d = dy->shape_view().Count(axis + 1);
    RUN_CUDA_KERNEL((L2NormalizeBackward<T>), ctx->stream(), n, n, c, d, static_cast<T>(epsilon),
                    y->dptr<T>(), dy->dptr<T>(), square_x_sum->dptr<T>(), dx->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_L2_NORMALIZE_GRAD_KERNEL(dtype)                  \
  REGISTER_USER_KERNEL("l2_normalize_grad")                            \
      .SetCreateFn<GpuL2NormalizeGradKernel<dtype>>()                  \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_CUDA_L2_NORMALIZE_GRAD_KERNEL(float)
REGISTER_CUDA_L2_NORMALIZE_GRAD_KERNEL(double)

}  // namespace oneflow
