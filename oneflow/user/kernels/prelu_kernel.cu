#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ndarray/ndarray_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void PReluForwardGpu(const int32_t elem_cnt, const int32_t alpha_size,
                                const int32_t inner_size, const T* x, const T* alpha, T* y) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const T x_i = x[i];
    const T alpha_i = alpha[(i / inner_size) % alpha_size];
    y[i] = x_i > 0 ? x_i : x_i * alpha_i;
  }
}

template<typename T>
__global__ void PReluBackwardGpu(const int32_t elem_cnt, const int32_t alpha_size,
                                 const int32_t inner_size, const T* x, const T* alpha, const T* dy,
                                 T* dx, T* alpha_diff) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const T x_i = x[i];
    const T dy_i = dy[i];
    const T alpha_i = alpha[(i / inner_size) % alpha_size];
    dx[i] = x_i > 0 ? dy_i : dy_i * alpha_i;
    alpha_diff[(i / inner_size) % alpha_size] += x_i > 0 ? 0 : dy_i * x_i;
  }
}

}  // namespace

template<typename T>
class GpuPReluKernel final : public user_op::OpKernel {
 public:
  GpuPReluKernel() = default;
  ~GpuPReluKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* alpha = ctx->Tensor4ArgNameAndIndex("alpha", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const int32_t elem_cnt = x->shape().elem_cnt();
    const int32_t alpha_size = alpha->shape().elem_cnt();
    const int batch = x->shape().At(0);
    const int channels = x->shape().At(1);
    const int32_t inner_size = elem_cnt / batch / channels;
    PReluForwardGpu<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                         ctx->device_ctx()->hip_stream()>>>(
        elem_cnt, alpha_size, inner_size, x->dptr<T>(), alpha->dptr<T>(), y->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GPU_PRELU_KERNEL(dtype)                                              \
  REGISTER_USER_KERNEL("prelu").SetCreateFn<GpuPReluKernel<dtype>>().SetIsMatchedHob( \
      (user_op::HobDeviceTag() == "gpu")                                              \
      & (user_op::HobDataType("y", 0) == GetDataType<dtype>::value));

REGISTER_GPU_PRELU_KERNEL(float)
REGISTER_GPU_PRELU_KERNEL(double)

template<typename T>
class GpuPReluGradKernel final : public user_op::OpKernel {
 public:
  GpuPReluGradKernel() = default;
  ~GpuPReluGradKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* alpha = ctx->Tensor4ArgNameAndIndex("alpha", 0);
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    user_op::Tensor* alpha_diff = ctx->Tensor4ArgNameAndIndex("alpha_diff", 0);

    const int32_t elem_cnt = x->shape().elem_cnt();
    const int32_t alpha_size = alpha->shape().elem_cnt();
    const int batch = x->shape().At(0);
    const int channels = x->shape().At(1);
    const int32_t inner_size = elem_cnt / batch / channels;

    PReluBackwardGpu<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                          ctx->device_ctx()->hip_stream()>>>(
        elem_cnt, alpha_size, inner_size, x->dptr<T>(), alpha->dptr<T>(), dy->dptr<T>(),
        dx->mut_dptr<T>(), alpha_diff->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GPU_PRELU_GRAD_KERNEL(dtype)             \
  REGISTER_USER_KERNEL("prelu_grad")                      \
      .SetCreateFn<GpuPReluGradKernel<dtype>>()           \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu") \
                       & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_GPU_PRELU_GRAD_KERNEL(float)
REGISTER_GPU_PRELU_GRAD_KERNEL(double)

}  // namespace oneflow
