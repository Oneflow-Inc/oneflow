#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ndarray/ndarray_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/elementwise.cuh"
namespace oneflow {

namespace {

constexpr int32_t kVecSize = 4;
constexpr int32_t kBlockSize = 256;

template<typename T>
constexpr int32_t GetPreluPackSize() {
  // For float, bfloat16, half.
  return 4;
};

template<>
constexpr int32_t GetPreluPackSize<half>() {
  return 8;
};

template<>
constexpr int32_t GetPreluPackSize<double>() {
  return 2;
};

#if CUDA_VERSION >= 11000
#define RETURN_VOID_IF_HALF                                                                        \
  typename std::enable_if_t<(std::is_same<T, half>::value || std::is_same<T, hip_bfloat16>::value), \
                            void>
#else
#define RETURN_VOID_IF_HALF typename std::enable_if_t<std::is_same<T, half>::value, void>
#endif
#define RETURN_VOID_IF_FLOAT typename std::enable_if_t<std::is_same<T, float>::value, void>
#define RETURN_VOID_IF_DOUBLE typename std::enable_if_t<std::is_same<T, double>::value, void>

template<typename T, typename IndexType, int pack_size, bool tail>
__global__ void PReluForwardMultiAlphaGpu(const IndexType elem_cnt, const IndexType alpha_size,
                                          const IndexType inner_size, const IndexType n_tail,
                                          const T* x, const T* alpha, T* y, const T* tail_x,
                                          T* tail_y) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  using LoadType = cuda::elementwise::PackType<T, pack_size>;
  using LoadPack = cuda::elementwise::Pack<T, pack_size>;
  T zero_val = static_cast<T>(0);
  for (int64_t linear_index = global_thread_id * pack_size; linear_index < elem_cnt;
       linear_index += gridDim.x * blockDim.x * pack_size) {
    const LoadType* x_load = reinterpret_cast<const LoadType*>(x + linear_index);
    LoadPack x_vec;
    x_vec.storage = *x_load;

    LoadPack y_vec;
#pragma unroll
    for (int i = 0; i < pack_size; i++) {
      y_vec.elem[i] = x_vec.elem[i] > zero_val
                          ? x_vec.elem[i]
                          : x_vec.elem[i] * alpha[(linear_index / inner_size) % alpha_size];
    }
    *(reinterpret_cast<LoadType*>(y + linear_index)) = y_vec.storage;
  }

  if (tail && global_thread_id < n_tail) {
    T tail_x_val = tail_x[global_thread_id];
    tail_y[global_thread_id] =
        tail_x_val > zero_val ? tail_x_val
                              : tail_x_val * alpha[(global_thread_id / inner_size) % alpha_size];
  }
}

template<typename T, typename IndexType, int pack_size, bool tail>
__global__ void PReluBackwardMultiAlphaGpu(const IndexType elem_cnt, const IndexType alpha_size,
                                           const IndexType inner_size, const IndexType n_tail,
                                           const T* x, const T* alpha, const T* dy, T* dx,
                                           T* alpha_diff, const T* tail_x, const T* tail_dy,
                                           T* tail_dx) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  using LoadType = cuda::elementwise::PackType<T, pack_size>;
  using LoadPack = cuda::elementwise::Pack<T, pack_size>;
  T zero_val = static_cast<T>(0);

  for (int64_t linear_index = global_thread_id * pack_size; linear_index < elem_cnt;
       linear_index += gridDim.x * blockDim.x * pack_size) {
    const LoadType* x_load = reinterpret_cast<const LoadType*>(x + linear_index);
    LoadPack x_vec;
    x_vec.storage = *x_load;

    const LoadType* dy_load = reinterpret_cast<const LoadType*>(dy + linear_index);
    LoadPack dy_vec;
    dy_vec.storage = *dy_load;

    LoadPack dx_vec;
#pragma unroll
    for (int i = 0; i < pack_size; i++) {
      dx_vec.elem[i] = x_vec.elem[i] > zero_val
                           ? dy_vec.elem[i]
                           : dy_vec.elem[i] * alpha[(linear_index / inner_size) % alpha_size];
      alpha_diff[(linear_index / inner_size) % alpha_size] +=
          x_vec.elem[i] > zero_val ? zero_val : dy_vec.elem[i] * x_vec.elem[i];
    }

    *(reinterpret_cast<LoadType*>(dx + linear_index)) = dx_vec.storage;
  }

  if (tail && global_thread_id < n_tail) {
    T tail_x_val = tail_x[global_thread_id];
    T tail_dy_val = tail_dy[global_thread_id];
    tail_dx[global_thread_id] =
        tail_x_val > zero_val ? tail_dy_val
                              : tail_dy_val * alpha[(global_thread_id / inner_size) % alpha_size];
    alpha_diff[(global_thread_id / inner_size) % alpha_size] +=
        tail_x_val > zero_val ? zero_val : tail_dy_val * tail_x[global_thread_id];
  }
}

template<typename T>
void DispatchPreluForwardIndexTail(ep::Stream* stream, const int64_t elem_cnt,
                                   const int64_t alpha_size, const int64_t inner_size, const T* x,
                                   const T* alpha, T* y) {
  constexpr int pack_size = GetPreluPackSize<T>();
  const int64_t pack_num = elem_cnt / pack_size;
  int grid_size;
  hipError_t err = cuda::elementwise::GetNumBlocks(pack_num, &grid_size);

  const int64_t tail_offset = pack_num * pack_size;
  const int64_t n_tail = elem_cnt - tail_offset;
  const bool tail = n_tail > 0 ? true : false;
  uint64_t inc_offset = 0;

  if (elem_cnt < GetMaxVal<int32_t>()) {
    if (tail) {
      // If tail, we need generate randnum one more time, so here we add another `1`.
      inc_offset = ((elem_cnt - 1) / (kBlockSize * grid_size * kVecSize) + 1) * kVecSize + 1;
      PReluForwardMultiAlphaGpu<T, int32_t, pack_size, true>
          <<<grid_size, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
              elem_cnt, alpha_size, inner_size, n_tail, x, alpha, y, (x + tail_offset),
              (y + tail_offset));
    } else {
      inc_offset = ((elem_cnt - 1) / (kBlockSize * grid_size * kVecSize) + 1) * kVecSize;
      PReluForwardMultiAlphaGpu<T, int32_t, pack_size, false>
          <<<grid_size, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
              elem_cnt, alpha_size, inner_size, n_tail, x, alpha, y, nullptr, nullptr);
    }
  } else {
    if (tail) {
      // If tail, we need generate randnum one more time, so here we add another `1`.
      inc_offset = ((elem_cnt - 1) / (kBlockSize * grid_size * kVecSize) + 1) * kVecSize + 1;
      PReluForwardMultiAlphaGpu<T, int64_t, pack_size, true>
          <<<grid_size, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
              elem_cnt, alpha_size, inner_size, n_tail, x, alpha, y, (x + tail_offset),
              (y + tail_offset));
    } else {
      inc_offset = ((elem_cnt - 1) / (kBlockSize * grid_size * kVecSize) + 1) * kVecSize;
      PReluForwardMultiAlphaGpu<T, int64_t, pack_size, false>
          <<<grid_size, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
              elem_cnt, alpha_size, inner_size, n_tail, x, alpha, y, nullptr, nullptr);
    }
  }
}

template<typename T>
void DispatchBackwardIndexTail(ep::Stream* stream, const int64_t elem_cnt, const int64_t alpha_size,
                               const int64_t inner_size, const T* x, const T* alpha, const T* dy,
                               T* dx, T* alpha_diff) {
  constexpr int pack_size = GetPreluPackSize<T>();
  const int64_t pack_num = elem_cnt / pack_size;
  int grid_size;
  hipError_t err = cuda::elementwise::GetNumBlocks(pack_num, &grid_size);
  const int64_t tail_offset = pack_num * pack_size;
  const int64_t n_tail = elem_cnt - tail_offset;
  const bool tail = n_tail > 0 ? true : false;
  uint64_t inc_offset = 0;

  if (elem_cnt < GetMaxVal<int32_t>()) {
    if (tail) {
      // If tail, we need generate randnum one more time, so here we add another `1`.
      PReluBackwardMultiAlphaGpu<T, int32_t, pack_size, true>
          <<<grid_size, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
              elem_cnt, alpha_size, inner_size, n_tail, x, alpha, dy, dx, alpha_diff,
              (x + tail_offset), (dy + tail_offset), (dx + tail_offset));
    } else {
      PReluBackwardMultiAlphaGpu<T, int32_t, pack_size, false>
          <<<grid_size, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
              elem_cnt, alpha_size, inner_size, n_tail, x, alpha, dy, dx, alpha_diff, nullptr,
              nullptr, nullptr);
    }
  } else {
    if (tail) {
      // If tail, we need generate randnum one more time, so here we add another `1`.
      PReluBackwardMultiAlphaGpu<T, int64_t, pack_size, true>
          <<<grid_size, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
              elem_cnt, alpha_size, inner_size, n_tail, x, alpha, dy, dx, alpha_diff,
              (x + tail_offset), (dy + tail_offset), (dx + tail_offset));
    } else {
      PReluBackwardMultiAlphaGpu<T, int64_t, pack_size, false>
          <<<grid_size, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
              elem_cnt, alpha_size, inner_size, n_tail, x, alpha, dy, dx, alpha_diff, nullptr,
              nullptr, nullptr);
    }
  }
}

template<typename T>
__global__ void PReluForwardSingleAlphaGpu(const int32_t elem_cnt, const T* x, const T* alpha,
                                           T* y) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) { y[i] = x[i] > 0 ? x[i] : x[i] * alpha[0]; }
}

template<typename T>
__global__ void PReluBackwardSingleAlphaGpu(const int32_t elem_cnt, const T* x, const T* alpha,
                                            const T* dy, T* dx, T* alpha_diff) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    dx[i] = x[i] > 0 ? dy[i] : dy[i] * alpha[0];
    alpha_diff[0] += x[i] > 0 ? 0 : dy[i] * x[i];
  }
}

template<>
__global__ void PReluForwardSingleAlphaGpu<half>(const int32_t elem_cnt, const half* x,
                                                 const half* alpha, half* y) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    y[i] = x[i] > static_cast<half>(0.0) ? x[i] : __hmul(x[i], alpha[0]);
  }
}

template<>
__global__ void PReluBackwardSingleAlphaGpu<half>(const int32_t elem_cnt, const half* x,
                                                  const half* alpha, const half* dy, half* dx,
                                                  half* alpha_diff) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    dx[i] = x[i] > static_cast<half>(0.0) ? dy[i] : __hmul(dy[i], alpha[0]);
    alpha_diff[0] = __hadd(alpha_diff[0], x[i] > static_cast<half>(0.0) ? static_cast<half>(0.0)
                                                                        : __hmul(dy[i], x[i]));
  }
}

}  // namespace

template<typename T>
class GpuPReluKernel final : public user_op::OpKernel {
 public:
  GpuPReluKernel() = default;
  ~GpuPReluKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* alpha = ctx->Tensor4ArgNameAndIndex("alpha", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const int64_t elem_cnt = x->shape().elem_cnt();
    const int64_t alpha_size = alpha->shape().elem_cnt();
    if (alpha_size == 1) {
      PReluForwardSingleAlphaGpu<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                      ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          elem_cnt, x->dptr<T>(), alpha->dptr<T>(), y->mut_dptr<T>());
    } else {
      const int64_t batch = x->shape().At(0);
      const int64_t channels = x->shape().At(1);
      const int64_t inner_size = elem_cnt / batch / channels;
      DispatchPreluForwardIndexTail<T>(
          ctx->stream(), elem_cnt, alpha_size, inner_size, reinterpret_cast<const T*>(x->dptr()),
          reinterpret_cast<const T*>(alpha->dptr()), reinterpret_cast<T*>(y->mut_dptr()));
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_PRELU_KERNEL(dtype)                                             \
  REGISTER_USER_KERNEL("prelu").SetCreateFn<GpuPReluKernel<dtype>>().SetIsMatchedHob( \
      (user_op::HobDeviceType() == DeviceType::kCUDA)                                 \
      && (user_op::HobDataType("y", 0) == GetDataType<dtype>::value));

REGISTER_CUDA_PRELU_KERNEL(float)
REGISTER_CUDA_PRELU_KERNEL(double)
REGISTER_CUDA_PRELU_KERNEL(half)

template<typename T>
class GpuPReluGradKernel final : public user_op::OpKernel {
 public:
  GpuPReluGradKernel() = default;
  ~GpuPReluGradKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* alpha = ctx->Tensor4ArgNameAndIndex("alpha", 0);
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    user_op::Tensor* alpha_diff = ctx->Tensor4ArgNameAndIndex("alpha_diff", 0);

    Memset<DeviceType::kCUDA>(ctx->stream(), alpha_diff->mut_dptr<T>(), 0,
                              alpha_diff->shape().elem_cnt() * sizeof(T));

    const int64_t elem_cnt = x->shape().elem_cnt();
    const int64_t alpha_size = alpha->shape().elem_cnt();

    if (alpha_size == 1) {
      PReluBackwardSingleAlphaGpu<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                       ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          elem_cnt, x->dptr<T>(), alpha->dptr<T>(), dy->dptr<T>(), dx->mut_dptr<T>(),
          alpha_diff->mut_dptr<T>());
    } else {
      const int64_t batch = x->shape().At(0);
      const int64_t channels = x->shape().At(1);
      const int64_t inner_size = elem_cnt / batch / channels;

      DispatchBackwardIndexTail<T>(
          ctx->stream(), x->shape().elem_cnt(), alpha_size, inner_size,
          reinterpret_cast<const T*>(x->dptr()), reinterpret_cast<const T*>(alpha->dptr()),
          reinterpret_cast<const T*>(dy->dptr()), reinterpret_cast<T*>(dx->mut_dptr()),
          reinterpret_cast<T*>(alpha_diff->mut_dptr()));
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_PRELU_GRAD_KERNEL(dtype)                         \
  REGISTER_USER_KERNEL("prelu_grad")                                   \
      .SetCreateFn<GpuPReluGradKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_CUDA_PRELU_GRAD_KERNEL(float)
REGISTER_CUDA_PRELU_GRAD_KERNEL(double)
REGISTER_CUDA_PRELU_GRAD_KERNEL(half)

}  // namespace oneflow
