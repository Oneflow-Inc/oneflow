#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/kernel_util.cuh"

#include <float.h>

namespace oneflow {

namespace {

// NOTE(Liang Depeng): refer to
// https://stackoverflow.com/questions/17371275/implementing-max-reduce-in-cuda
template<typename T>
__global__ void ReduceMaxMinPerLayer(const T *input_ptr, const int64_t elements, T *max_ptr,
                                     T *min_ptr) {
  extern __shared__ unsigned char shared_max_min_memory[];
  T *shared_max = reinterpret_cast<T *>(shared_max_min_memory);
  T *shared_min = shared_max + blockDim.x;

  int64_t tid = threadIdx.x;
  int64_t gid = (blockDim.x * blockIdx.x) + tid;
  shared_max[tid] = -FLT_MAX;
  shared_min[tid] = -FLT_MAX;

  while (gid < elements) {
    shared_max[tid] = max(shared_max[tid], input_ptr[gid]);
    shared_min[tid] = max(shared_min[tid], -input_ptr[gid]);
    gid += gridDim.x * blockDim.x;
  }
  __syncthreads();
  gid = (blockDim.x * blockIdx.x) + tid;
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s && gid < elements) {
      shared_max[tid] = max(shared_max[tid], shared_max[tid + s]);
      shared_min[tid] = max(shared_min[tid], shared_min[tid + s]);
    }
    __syncthreads();
  }

  if (tid == 0) {
    gpu_atomic_max(max_ptr, shared_max[0]);
    gpu_atomic_max(min_ptr, shared_min[0]);
  }
}

template<typename T>
__global__ void InitMaxMin(const int64_t elements, T *max_ptr, T *min_ptr) {
  int64_t tid = threadIdx.x;
  int64_t gid = (blockDim.x * blockIdx.x) + tid;

  while (gid < elements) {
    max_ptr[gid] = -FLT_MAX;
    min_ptr[gid] = -FLT_MAX;
    gid += gridDim.x * blockDim.x;
  }
}

template<typename T>
__global__ void CalScaleZeroPointSymmetric(const int64_t elements, const double quantization_bit,
                                           const float momentum, const T *max_ptr, const T *min_ptr,
                                           T *moving_max_ptr, T *moving_min_ptr, T *scale,
                                           T *zero_point) {
  int64_t tid = threadIdx.x;
  int64_t gid = (blockDim.x * blockIdx.x) + tid;

  while (gid < elements) {
    T activation_max = max(fabs(max_ptr[gid]), fabs(min_ptr[gid]));
    T denominator = static_cast<T>(pow(2.0, quantization_bit - 1)) - 1;

    if (moving_max_ptr[gid] == 0)
      moving_max_ptr[gid] = activation_max;
    else
      moving_max_ptr[gid] = moving_max_ptr[gid] * momentum + activation_max * (1 - momentum);

    // NOTE(Liang Depeng): symmetric quantization only use moving_max to calculate the scale
    moving_min_ptr[gid] = moving_max_ptr[gid];

    scale[gid] = moving_max_ptr[gid] / denominator;
    zero_point[gid] = 0;
    gid += gridDim.x * blockDim.x;
  }
}

template<typename T>
__global__ void CalFreezeScaleZeroPointSymmetric(const int64_t elements,
                                                 const double quantization_bit,
                                                 const float momentum, const T *moving_max_ptr,
                                                 T *scale, T *zero_point) {
  int64_t tid = threadIdx.x;
  int64_t gid = (blockDim.x * blockIdx.x) + tid;

  while (gid < elements) {
    T denominator = static_cast<T>(pow(2.0, quantization_bit - 1)) - 1;
    scale[gid] = moving_max_ptr[gid] / denominator;
    zero_point[gid] = 0;
    gid += gridDim.x * blockDim.x;
  }
}

template<typename T>
__global__ void CalScaleZeroPointAffine(const int64_t elements, const double quantization_bit,
                                        const float momentum, const T *max_ptr, const T *min_ptr,
                                        T *moving_max_ptr, T *moving_min_ptr, T *scale,
                                        T *zero_point) {
  int64_t tid = threadIdx.x;
  int64_t gid = (blockDim.x * blockIdx.x) + tid;

  while (gid < elements) {
    T denominator = static_cast<T>(pow(2.0, quantization_bit)) - 1;

    if (moving_max_ptr[gid] == 0)
      moving_max_ptr[gid] = max_ptr[gid];
    else
      moving_max_ptr[gid] = moving_max_ptr[gid] * momentum + max_ptr[gid] * (1 - momentum);

    if (moving_min_ptr[gid] == 0)
      moving_min_ptr[gid] = -min_ptr[gid];
    else
      moving_min_ptr[gid] = moving_min_ptr[gid] * momentum + -min_ptr[gid] * (1 - momentum);

    T min = moving_min_ptr[gid];
    T s = (moving_max_ptr[gid] - min) / denominator;

    scale[gid] = s;
    zero_point[gid] = -min / s;
    gid += gridDim.x * blockDim.x;
  }
}

template<typename T>
__global__ void CalFreezeScaleZeroPointAffine(const int64_t elements, const double quantization_bit,
                                              const float momentum, const T *moving_max_ptr,
                                              const T *moving_min_ptr, T *scale, T *zero_point) {
  int64_t tid = threadIdx.x;
  int64_t gid = (blockDim.x * blockIdx.x) + tid;

  while (gid < elements) {
    T denominator = static_cast<T>(pow(2.0, quantization_bit)) - 1;

    T min = moving_min_ptr[gid];
    T s = (moving_max_ptr[gid] - min) / denominator;

    scale[gid] = s;
    zero_point[gid] = -min / s;
    gid += gridDim.x * blockDim.x;
  }
}

}  // namespace

#define LAUNCH_CUDA_KERNEL(func, device_ctx_ptr, thread_num, shared_mem_size, ...)     \
  func<<<SMBlocksNum4ThreadsNum(thread_num), kCudaThreadsNumPerBlock, shared_mem_size, \
         (device_ctx_ptr)->hip_stream()>>>(__VA_ARGS__)

template<typename T>
class GpuMovingAverageMinMaxObserverKernel final : public user_op::OpKernel {
 public:
  GpuMovingAverageMinMaxObserverKernel() = default;
  ~GpuMovingAverageMinMaxObserverKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext *ctx) const override {
    const user_op::Tensor *in = ctx->Tensor4ArgNameAndIndex("in", 0);
    const user_op::Tensor *current_train_step =
        ctx->Tensor4ArgNameAndIndex("current_train_step", 0);
    user_op::Tensor *moving_max = ctx->Tensor4ArgNameAndIndex("moving_max", 0);
    user_op::Tensor *moving_min = ctx->Tensor4ArgNameAndIndex("moving_min", 0);
    user_op::Tensor *scale = ctx->Tensor4ArgNameAndIndex("scale", 0);
    user_op::Tensor *zero_point = ctx->Tensor4ArgNameAndIndex("zero_point", 0);
    user_op::Tensor *tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    const bool is_training = ctx->Attr<bool>("training");
    const int64_t stop_update_after_iters = ctx->Attr<int64_t>("stop_update_after_iters");
    const std::string quantization_scheme = ctx->Attr<std::string>("quantization_scheme");
    const int32_t quantization_bit = ctx->Attr<int32_t>("quantization_bit");
    const float momentum = ctx->Attr<float>("momentum");

    int64_t elements = in->shape().elem_cnt();
    T *max_ptr = tmp_buffer->mut_dptr<T>();
    T *min_ptr = max_ptr + 1;

    int64_t *host_current_train_step_ptr = new int64_t[current_train_step->shape().elem_cnt()];
    OF_CUDA_CHECK(hipMemcpy(host_current_train_step_ptr, current_train_step->dptr<int64_t>(),
                             current_train_step->shape().elem_cnt() * sizeof(int64_t),
                             hipMemcpyDefault));

    if (*host_current_train_step_ptr <= stop_update_after_iters && is_training) {
      LAUNCH_CUDA_KERNEL((InitMaxMin<T>), ctx->device_ctx(), 1, 0, 1, max_ptr, min_ptr);
      LAUNCH_CUDA_KERNEL((ReduceMaxMinPerLayer<T>), ctx->device_ctx(), elements,
                         kCudaThreadsNumPerBlock * 2 * sizeof(T), in->dptr<T>(), elements, max_ptr,
                         min_ptr);
    }

    if (quantization_scheme == "symmetric") {
      if (*host_current_train_step_ptr <= stop_update_after_iters) {
        LAUNCH_CUDA_KERNEL((CalScaleZeroPointSymmetric<T>), ctx->device_ctx(), 1, 0, 1,
                           static_cast<double>(quantization_bit), momentum, max_ptr, min_ptr,
                           moving_max->mut_dptr<T>(), moving_min->mut_dptr<T>(),
                           scale->mut_dptr<T>(), zero_point->mut_dptr<T>());
      } else {
        LAUNCH_CUDA_KERNEL((CalFreezeScaleZeroPointSymmetric<T>), ctx->device_ctx(), 1, 0, 1,
                           static_cast<double>(quantization_bit), momentum, moving_max->dptr<T>(),
                           scale->mut_dptr<T>(), zero_point->mut_dptr<T>());
      }
    } else {  // quantization_scheme == "affine"
      if (*host_current_train_step_ptr <= stop_update_after_iters) {
        LAUNCH_CUDA_KERNEL((CalScaleZeroPointAffine<T>), ctx->device_ctx(), 1, 0, 1,
                           static_cast<double>(quantization_bit), momentum, max_ptr, min_ptr,
                           moving_max->mut_dptr<T>(), moving_min->mut_dptr<T>(),
                           scale->mut_dptr<T>(), zero_point->mut_dptr<T>());
      } else {
        LAUNCH_CUDA_KERNEL((CalFreezeScaleZeroPointAffine<T>), ctx->device_ctx(), 1, 0, 1,
                           static_cast<double>(quantization_bit), momentum, moving_max->dptr<T>(),
                           moving_min->dptr<T>(), scale->mut_dptr<T>(), zero_point->mut_dptr<T>());
      }
    }

    delete[] host_current_train_step_ptr;
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_MOVING_AVERAGE_MIN_MAX_OBSERVER_KERNEL(dtype)                         \
  REGISTER_USER_KERNEL("moving_average_min_max_observer")                              \
      .SetCreateFn<GpuMovingAverageMinMaxObserverKernel<dtype>>()                      \
      .SetIsMatchedHob((user_op::HobDeviceTag() == DeviceType::kGPU)                   \
                       & (user_op::HobDataType("in", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn([](user_op::InferContext *ctx) -> size_t { return 2 * sizeof(dtype); })

REGISTER_MOVING_AVERAGE_MIN_MAX_OBSERVER_KERNEL(float);
REGISTER_MOVING_AVERAGE_MIN_MAX_OBSERVER_KERNEL(double);

}  // namespace oneflow
