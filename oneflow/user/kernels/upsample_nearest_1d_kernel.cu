#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/common/nd_index_offset_helper.h"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/user/kernels/upsample_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void UpsampleNearest1DForward(const int64_t elem_cnt, const T* in_dptr,
                                         NdIndexOffsetHelper<int64_t, 3> in_helper,
                                         NdIndexOffsetHelper<int64_t, 3> out_helper,
                                         const int64_t in_height, const float scale_factor,
                                         T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h;
    out_helper.OffsetToNdIndex(index, n, c, h);
    const int64_t in_h = GetNearestInputIndex(h, scale_factor, in_height);
    out_dptr[index] = in_dptr[in_helper.NdIndexToOffset(n, c, in_h)];
  }
}

template<typename T>
__global__ void UpsampleNearest1DBackward(const int64_t elem_cnt, const T* dy_dptr,
                                          NdIndexOffsetHelper<int64_t, 3> dy_helper,
                                          NdIndexOffsetHelper<int64_t, 3> dx_helper,
                                          const int64_t in_height, const float scale_factor,
                                          T* dx_dptr) {
  CUDA_1D_KERNEL_LOOP(index, elem_cnt) {
    int64_t n, c, h;
    dy_helper.OffsetToNdIndex(index, n, c, h);
    const int64_t dx_h = GetNearestInputIndex(h, scale_factor, in_height);
    cuda::atomic::Add(dx_dptr + dx_helper.NdIndexToOffset(n, c, dx_h), dy_dptr[index]);
  }
}

}  // namespace

template<typename T>
class UpsampleNearest1DGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearest1DGPUKernel() = default;
  ~UpsampleNearest1DGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_blob = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_blob = ctx->Tensor4ArgNameAndIndex("y", 0);
    const float height_scale = ctx->Attr<float>("scale_factor");
    const int64_t elem_cnt = y_blob->shape().elem_cnt();
    const int64_t in_height = x_blob->shape().At(2);
    const int64_t out_height = y_blob->shape().At(2);
    if (in_height == out_height) {
      Memcpy<DeviceType::kGPU>(ctx->device_ctx(), y_blob->mut_dptr<void>(), x_blob->dptr<void>(),
                               x_blob->shape().elem_cnt() * GetSizeOfDataType(x_blob->data_type()));
    } else {
      NdIndexOffsetHelper<int64_t, 3> in_helper(x_blob->shape().At(0), x_blob->shape().At(1),
                                                x_blob->shape().At(2));
      NdIndexOffsetHelper<int64_t, 3> out_helper(y_blob->shape().At(0), y_blob->shape().At(1),
                                                 y_blob->shape().At(2));
      RUN_CUDA_KERNEL((UpsampleNearest1DForward<T>), ctx->device_ctx(), elem_cnt, elem_cnt,
                      x_blob->dptr<T>(), in_helper, out_helper, x_blob->shape().At(2),
                      1.f / height_scale, y_blob->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class UpsampleNearestGrad1DGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearestGrad1DGPUKernel() = default;
  ~UpsampleNearestGrad1DGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_blob = ctx->Tensor4ArgNameAndIndex("dx", 0);
    if (dx_blob == nullptr) { return; }
    Memset<DeviceType::kGPU>(ctx->device_ctx(), dx_blob->mut_dptr<T>(), 0,
                             dx_blob->shape().elem_cnt() * sizeof(T));
    const user_op::Tensor* dy_blob = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const float height_scale = ctx->Attr<float>("scale_factor");
    const int64_t elem_cnt = dy_blob->shape().elem_cnt();
    const int64_t in_height = dx_blob->shape().At(2);
    const int64_t out_height = dy_blob->shape().At(2);
    if (in_height == out_height) {
      Memcpy<DeviceType::kGPU>(
          ctx->device_ctx(), dx_blob->mut_dptr<void>(), dy_blob->dptr<void>(),
          dy_blob->shape().elem_cnt() * GetSizeOfDataType(dy_blob->data_type()));
    } else {
      NdIndexOffsetHelper<int64_t, 3> dy_helper(dy_blob->shape().At(0), dy_blob->shape().At(1),
                                                dy_blob->shape().At(2));
      NdIndexOffsetHelper<int64_t, 3> dx_helper(dx_blob->shape().At(0), dx_blob->shape().At(1),
                                                dx_blob->shape().At(2));
      RUN_CUDA_KERNEL((UpsampleNearest1DBackward<T>), ctx->device_ctx(), elem_cnt, elem_cnt,
                      dy_blob->dptr<T>(), dy_helper, dx_helper, dx_blob->shape().At(2),
                      1.f / height_scale, dx_blob->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_UPSAMPNEAREST1D_GPU_KERNEL(dtype)                                     \
  REGISTER_USER_KERNEL("upsample_nearest_1d")                                          \
      .SetCreateFn<UpsampleNearest1DGPUKernel<dtype>>()                                \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                              \
                       & (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)); \
  REGISTER_USER_KERNEL("upsample_nearest_1d_grad")                                     \
      .SetCreateFn<UpsampleNearestGrad1DGPUKernel<dtype>>()                            \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                              \
                       & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_UPSAMPNEAREST1D_GPU_KERNEL(float)
REGISTER_UPSAMPNEAREST1D_GPU_KERNEL(double)

}  // namespace oneflow
