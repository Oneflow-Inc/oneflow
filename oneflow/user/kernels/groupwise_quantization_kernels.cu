#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include <hipcub/hipcub.hpp>

namespace oneflow {

namespace {

template<typename T, int pack_size>
struct alignas(sizeof(T) * pack_size) AlignedArray {
  __device__ AlignedArray() {
    // do nothing
  }
  union {
    T elem[pack_size];
  };
};

template<typename Src, typename Dst, size_t pack_size>
struct Cast {
  __device__ void operator()(const AlignedArray<Src, pack_size>& src,
                             AlignedArray<Dst, pack_size>* dst) {
#pragma unroll
    for (int i = 0; i < pack_size; ++i) { dst->elem[i] = static_cast<Dst>(src.elem[i]); }
  }
};

template<typename Dst, size_t pack_size>
struct Cast<uint8_t, Dst, pack_size> {
  __device__ void operator()(const AlignedArray<uint8_t, pack_size>& src,
                             AlignedArray<Dst, pack_size>* dst) {
#pragma unroll
    for (int i = 0; i < pack_size; ++i) { dst->elem[i] = static_cast<Dst>(src.elem[i]); }
  }

  __device__ void operator()(const AlignedArray<uint8_t, pack_size>& src,
                             AlignedArray<Dst, pack_size * 2>* dst) {
#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      const uint8_t q = src.elem[i];
      const uint8_t hi = (q >> 4);
      const uint8_t lo = (q & 0xF);
      dst->elem[i * 2 + 0] = static_cast<Dst>(hi);
      dst->elem[i * 2 + 1] = static_cast<Dst>(lo);
    }
  }
};

template<typename Dst, size_t pack_size>
struct Cast<int8_t, Dst, pack_size> {
  __device__ void operator()(const AlignedArray<int8_t, pack_size>& src,
                             AlignedArray<Dst, pack_size>* dst) {
#pragma unroll
    for (int i = 0; i < pack_size; ++i) { dst->elem[i] = static_cast<Dst>(src.elem[i]); }
  }

  __device__ void operator()(const AlignedArray<int8_t, pack_size>& src,
                             AlignedArray<Dst, pack_size * 2>* dst) {
#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      const int8_t q = src.elem[i];
      const int8_t hi = (q >> 4);
      int8_t lo = (q << 4);
      lo = (lo >> 4);
      dst->elem[i * 2 + 0] = static_cast<Dst>(hi);
      dst->elem[i * 2 + 1] = static_cast<Dst>(lo);
    }
  }
};

template<typename C, size_t pack_size>
struct InplaceAddScalar {
  __device__ void operator()(AlignedArray<C, pack_size>* array, C scalar) {
#pragma unroll
    for (int i = 0; i < pack_size; ++i) { array->elem[i] += scalar; }
  }
};

template<typename T, size_t pack_size>
struct InplaceFmaScalar {
  __device__ void operator()(AlignedArray<T, pack_size>* array, T m, T a) {
#pragma unroll
    for (int i = 0; i < pack_size; ++i) { array->elem[i] = array->elem[i] * m + a; }
  }
};

#if __CUDA_ARCH_ >= 530
template<size_t pack_size>
struct InplaceFmaScalar<half, pack_size> {
  __device__ void operator()(AlignedArray<half, pack_size>* array, half m, half a) {
    if (pack_size == 1) {
#pragma unroll
      for (int i = 0; i < pack_size; ++i) { array->elem[i] = array->elem[i] * m + a; }
    } else {
      const half2 m2 = __half2half2(m);
      const half2 a2 = __half2half2(a);
      half2* h2 = reinterpret_cast<half2*>(array->elem);
#pragma unroll
      for (int i = 0; i < pack_size / 2; ++i) { h2[i] = __hfma2(h2[i], m2, a2); }
    }
  }
};
#endif  // __CUDA_ARCH_ >= 530

template<typename T, size_t pack_size>
struct InplaceFma {
  __device__ void operator()(AlignedArray<T, pack_size>* a, const AlignedArray<T, pack_size>& b,
                             const AlignedArray<T, pack_size>& c) {
#pragma unroll
    for (int i = 0; i < pack_size; ++i) { a->elem[i] = a->elem[i] * b.elem[i] + c.elem[i]; }
  }
};

template<typename T, size_t pack_size>
struct InplaceMulScalar {
  __device__ void operator()(AlignedArray<T, pack_size>* a, T b) {
#pragma unroll
    for (int i = 0; i < pack_size; ++i) { a->elem[i] = a->elem[i] * b; }
  }
};

template<typename T, typename C, size_t pack_size>
struct MultiplyAccumulate {
  __device__ void operator()(const AlignedArray<T, pack_size>& a,
                             const AlignedArray<T, pack_size>& b, C* sum) {
#pragma unroll
    for (int i = 0; i < pack_size; ++i) { *sum += static_cast<C>(a.elem[i] * b.elem[i]); }
  }
};

template<size_t pack_size>
struct MultiplyAccumulate<half, float, pack_size> {
  __device__ void operator()(const AlignedArray<half, pack_size>& a,
                             const AlignedArray<half, pack_size>& b, float* sum) {
    if (pack_size == 1) {
#pragma unroll
      for (int i = 0; i < pack_size; ++i) { *sum += static_cast<float>(a.elem[i] * b.elem[i]); }
    } else {
      const half2* a2 = reinterpret_cast<const half2*>(a.elem);
      const half2* b2 = reinterpret_cast<const half2*>(b.elem);

      for (int i = 0; i < pack_size / 2; ++i) {
        const half2 c2 = __hmul2(a2[i], b2[i]);
        const float2 f2 = __half22float2(c2);
        *sum += f2.x;
        *sum += f2.y;
      }
    }
  }
};

template<typename T, typename U, typename Index, size_t d_pack_size, size_t q_pack_size, int bits,
         bool symmetric, bool outer_size_1>
__global__ void Dequantize3D(Index packed_elem_cnt, Index group_size, Index packed_inner_size,
                             const AlignedArray<U, q_pack_size>* quantized,
                             const AlignedArray<T, d_pack_size>* scale,
                             const AlignedArray<T, d_pack_size>* zero,
                             AlignedArray<T, d_pack_size>* out) {
  const Index packed_group_inner_size = group_size * packed_inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, packed_elem_cnt) {
    const Index outer_id = outer_size_1 ? 0 : i / packed_group_inner_size;
    const Index group_inner_offset = i - outer_id * packed_group_inner_size;
    const Index group_id = group_inner_offset / packed_inner_size;
    const Index inner_id = group_inner_offset - group_id * packed_inner_size;
    const Index scale_offset = outer_id * packed_inner_size + inner_id;
    const AlignedArray<T, d_pack_size> group_scale = scale[scale_offset];
    AlignedArray<T, d_pack_size> group_zero;
    if (symmetric) {
      if (std::is_same<U, uint8_t>::value) {
        group_zero = group_scale;
        InplaceMulScalar<T, d_pack_size>()(&group_zero, -static_cast<T>(((1 << (bits - 1)) - 1)));
      } else {
#pragma unroll
        for (int i = 0; i < d_pack_size; ++i) { group_zero.elem[i] = 0; }
      }
    } else {
      group_zero = zero[scale_offset];
    }
    AlignedArray<T, d_pack_size> values;
    const AlignedArray<U, q_pack_size> q = quantized[i];
    Cast<U, T, q_pack_size>()(q, &values);
    InplaceFma<T, d_pack_size>()(&values, group_scale, group_zero);
    out[i] = values;
  }
}

template<typename T, typename U, int num_bits, bool symmetric, size_t d_pack_size,
         size_t q_pack_size, bool outer_size_1>
void LaunchDequantize3D(ep::CudaStream* stream, int64_t outer_size, int64_t group_size,
                        int64_t inner_size, const U* in, const T* scale, const T* zero, T* out) {
  if constexpr (sizeof(T) * d_pack_size <= 16 && q_pack_size > 0) {
    const int64_t packed_elem_cnt = outer_size * group_size * inner_size / d_pack_size;
    const int64_t packed_inner_size = inner_size / d_pack_size;
    if (packed_elem_cnt <= (1 << 30)) {
      RUN_CUDA_KERNEL((Dequantize3D<T, U, int32_t, d_pack_size, q_pack_size, num_bits, symmetric,
                                    outer_size_1>),
                      stream, packed_elem_cnt, packed_elem_cnt, group_size, packed_inner_size,
                      reinterpret_cast<const AlignedArray<U, q_pack_size>*>(in),
                      reinterpret_cast<const AlignedArray<T, d_pack_size>*>(scale),
                      reinterpret_cast<const AlignedArray<T, d_pack_size>*>(zero),
                      reinterpret_cast<AlignedArray<T, d_pack_size>*>(out));
    } else {
      RUN_CUDA_KERNEL((Dequantize3D<T, U, int64_t, d_pack_size, q_pack_size, num_bits, symmetric,
                                    outer_size_1>),
                      stream, packed_elem_cnt, packed_elem_cnt, group_size, packed_inner_size,
                      reinterpret_cast<const AlignedArray<U, q_pack_size>*>(in),
                      reinterpret_cast<const AlignedArray<T, d_pack_size>*>(scale),
                      reinterpret_cast<const AlignedArray<T, d_pack_size>*>(zero),
                      reinterpret_cast<AlignedArray<T, d_pack_size>*>(out));
    }
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T, typename U, int num_bits, bool symmetric, size_t d_pack_size,
         size_t q_pack_size>
void DispatchDequantize3DOuterSize1(ep::CudaStream* stream, int64_t outer_size, int64_t group_size,
                                    int64_t inner_size, const U* in, const T* scale, const T* zero,
                                    T* out) {
  if (outer_size == 1) {
    LaunchDequantize3D<T, U, num_bits, symmetric, d_pack_size, q_pack_size, true>(
        stream, outer_size, group_size, inner_size, in, scale, zero, out);
  } else {
    LaunchDequantize3D<T, U, num_bits, symmetric, d_pack_size, q_pack_size, false>(
        stream, outer_size, group_size, inner_size, in, scale, zero, out);
  }
}

template<typename T, typename U, int num_bits, bool symmetric>
void DispatchDequantize3D(ep::CudaStream* stream, int64_t outer_size, int64_t group_size,
                          int64_t inner_size, const U* in, const T* scale, const T* zero, T* out) {
  constexpr int32_t max_pack_size = 16 / sizeof(T);
  constexpr int32_t data_per_quant = 8 / num_bits;
  int32_t pack_size = max_pack_size;
  while (inner_size % pack_size != 0) { pack_size /= 2; }
  if (pack_size == 16) {
    DispatchDequantize3DOuterSize1<T, U, num_bits, symmetric, 16, 16 / data_per_quant>(
        stream, outer_size, group_size, inner_size, in, scale, zero, out);
  } else if (pack_size == 8) {
    DispatchDequantize3DOuterSize1<T, U, num_bits, symmetric, 8, 8 / data_per_quant>(
        stream, outer_size, group_size, inner_size, in, scale, zero, out);
  } else if (pack_size == 4) {
    DispatchDequantize3DOuterSize1<T, U, num_bits, symmetric, 4, 4 / data_per_quant>(
        stream, outer_size, group_size, inner_size, in, scale, zero, out);
  } else if (pack_size == 2) {
    DispatchDequantize3DOuterSize1<T, U, num_bits, symmetric, 2, 2 / data_per_quant>(
        stream, outer_size, group_size, inner_size, in, scale, zero, out);
  } else if (pack_size == 1) {
    DispatchDequantize3DOuterSize1<T, U, num_bits, symmetric, 1, 1 / data_per_quant>(
        stream, outer_size, group_size, inner_size, in, scale, zero, out);
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T, typename U, typename Index, size_t d_pack_size, size_t q_pack_size, int bits,
         bool symmetric>
__global__ void DequantizeInnerSize1(Index packed_elem_cnt, Index packed_group_size,
                                     const AlignedArray<U, q_pack_size>* quantized, const T* scale,
                                     const T* zero, AlignedArray<T, d_pack_size>* out) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, packed_elem_cnt) {
    const Index group_id = i / packed_group_size;
    const T group_scale = scale[group_id];
    T group_zero;
    if (symmetric) {
      if (std::is_same<U, uint8_t>::value) {
        group_zero = -static_cast<T>(((1 << (bits - 1)) - 1)) * group_scale;
      } else {
        group_zero = 0;
      }
    } else {
      group_zero = zero[group_id];
    }
    AlignedArray<T, d_pack_size> values;
    AlignedArray<U, q_pack_size> q = quantized[i];
    Cast<U, T, q_pack_size>()(q, &values);
    InplaceFmaScalar<T, d_pack_size>()(&values, group_scale, group_zero);
    out[i] = values;
  }
}

template<typename T, typename U, int num_bits, bool symmetric, size_t d_pack_size,
         size_t q_pack_size>
void LaunchDequantizeInnerSize1(ep::CudaStream* stream, int64_t outer_size, int64_t group_size,
                                const U* in, const T* scale, const T* zero, T* out) {
  if constexpr (sizeof(T) * d_pack_size <= 16 && q_pack_size > 0) {
    const int64_t packed_elem_cnt = outer_size * group_size / d_pack_size;
    const int64_t packed_group_size = group_size / d_pack_size;
    if (packed_elem_cnt <= (1 << 30)) {
      RUN_CUDA_KERNEL(
          (DequantizeInnerSize1<T, U, int32_t, d_pack_size, q_pack_size, num_bits, symmetric>),
          stream, packed_elem_cnt, packed_elem_cnt, packed_group_size,
          reinterpret_cast<const AlignedArray<U, q_pack_size>*>(in), scale, zero,
          reinterpret_cast<AlignedArray<T, d_pack_size>*>(out));
    } else {
      RUN_CUDA_KERNEL(
          (DequantizeInnerSize1<T, U, int64_t, d_pack_size, q_pack_size, num_bits, symmetric>),
          stream, packed_elem_cnt, packed_elem_cnt, packed_group_size,
          reinterpret_cast<const AlignedArray<U, q_pack_size>*>(in), scale, zero,
          reinterpret_cast<AlignedArray<T, d_pack_size>*>(out));
    }
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T, typename U, int num_bits, bool symmetric>
void DispatchDequantizeInnerSize1PackSize(ep::CudaStream* stream, int64_t outer_size,
                                          int64_t group_size, const U* in, const T* scale,
                                          const T* zero, T* out) {
  constexpr int32_t max_pack_size = 16 / sizeof(T);
  int32_t pack_size = max_pack_size;
  while (group_size % pack_size != 0) { pack_size /= 2; }
  constexpr int32_t data_per_quant = 8 / num_bits;
  CHECK(group_size % data_per_quant == 0);
  if (pack_size == 16) {
    LaunchDequantizeInnerSize1<T, U, num_bits, symmetric, 16, 16 / data_per_quant>(
        stream, outer_size, group_size, in, scale, zero, out);
  } else if (pack_size == 8) {
    LaunchDequantizeInnerSize1<T, U, num_bits, symmetric, 8, 8 / data_per_quant>(
        stream, outer_size, group_size, in, scale, zero, out);
  } else if (pack_size == 4) {
    LaunchDequantizeInnerSize1<T, U, num_bits, symmetric, 4, 4 / data_per_quant>(
        stream, outer_size, group_size, in, scale, zero, out);
  } else if (pack_size == 2) {
    LaunchDequantizeInnerSize1<T, U, num_bits, symmetric, 2, 2 / data_per_quant>(
        stream, outer_size, group_size, in, scale, zero, out);
  } else if (pack_size == 1) {
    LaunchDequantizeInnerSize1<T, U, num_bits, symmetric, 1, 1 / data_per_quant>(
        stream, outer_size, group_size, in, scale, zero, out);
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T, typename U, int num_bits, bool symmetric>
void DispatchDequantizeSize(ep::CudaStream* stream, int64_t outer_size, int64_t group_size,
                            int64_t inner_size, const U* in, const T* scale, const T* zero,
                            T* out) {
  if (inner_size == 1) {
    DispatchDequantizeInnerSize1PackSize<T, U, num_bits, symmetric>(stream, outer_size, group_size,
                                                                    in, scale, zero, out);
  } else {
    DispatchDequantize3D<T, U, num_bits, symmetric>(stream, outer_size, group_size, inner_size, in,
                                                    scale, zero, out);
  }
}

template<typename T, typename U>
void DispatchDequantize(ep::CudaStream* stream, int32_t num_bits, bool symmetric,
                        int64_t outer_size, int64_t group_size, int64_t inner_size, const U* in,
                        const T* scale, const T* zero, T* out) {
  if (num_bits == 4) {
    if (symmetric) {
      DispatchDequantizeSize<T, U, 4, true>(stream, outer_size, group_size, inner_size, in, scale,
                                            zero, out);
    } else {
      DispatchDequantizeSize<T, U, 4, false>(stream, outer_size, group_size, inner_size, in, scale,
                                             zero, out);
    }
  } else if (num_bits == 8) {
    if (symmetric) {
      DispatchDequantizeSize<T, U, 8, true>(stream, outer_size, group_size, inner_size, in, scale,
                                            zero, out);
    } else {
      DispatchDequantizeSize<T, U, 8, false>(stream, outer_size, group_size, inner_size, in, scale,
                                             zero, out);
    }

  } else {
    UNIMPLEMENTED();
  }
}

template<typename T>
class GroupwiseDequantizeKernel final : public user_op::OpKernel {
 public:
  GroupwiseDequantizeKernel() = default;
  ~GroupwiseDequantizeKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    const user_op::Tensor* scale = ctx->Tensor4ArgNameAndIndex("scale", 0);
    const user_op::Tensor* zero = nullptr;
    if (ctx->has_input("zero", 0)) { zero = ctx->Tensor4ArgNameAndIndex("zero", 0); }
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int64_t group_size = ctx->Attr<int64_t>("group_size");
    const int64_t group_dim = ctx->Attr<int64_t>("group_dim");
    const int32_t num_bits = ctx->Attr<int32_t>("num_bits");
    const bool symmetric = ctx->Attr<bool>("symmetric");
    const int64_t num_in_axes = in->shape_view().NumAxes();
    CHECK_GE(num_in_axes, 1);
    CHECK_EQ(scale->shape_view().NumAxes(), num_in_axes);
    if (zero != nullptr) { CHECK_EQ(zero->shape_view().NumAxes(), num_in_axes); }
    CHECK_EQ(out->shape_view().NumAxes(), num_in_axes);
    CHECK_GE(group_dim, 0);
    CHECK_LT(group_dim, num_in_axes);
    for (int i = 0; i < num_in_axes; ++i) {
      if (i == num_in_axes - 1) {
        CHECK_EQ(out->shape_view().At(i), in->shape_view().At(i) * (8 / num_bits));
      } else {
        CHECK_EQ(out->shape_view().At(i), in->shape_view().At(i));
      }
    }
    const int64_t group_dim_size = out->shape_view().At(group_dim);
    CHECK_GT(group_size, 0);
    CHECK_LE(group_size, group_dim_size);
    CHECK_EQ(group_dim_size % group_size, 0);
    const int64_t num_groups = group_dim_size / group_size;
    for (int i = 0; i < num_in_axes; ++i) {
      const int64_t expected_dim_size = i == group_dim ? num_groups : out->shape_view().At(i);
      CHECK_EQ(scale->shape_view().At(i), expected_dim_size);
      if (zero != nullptr) { CHECK_EQ(zero->shape_view().At(i), expected_dim_size); }
    }
    const int64_t outer_size = out->shape_view().Count(0, group_dim) * num_groups;
    const int64_t inner_size = out->shape_view().Count(group_dim + 1);
    if (in->data_type() == DataType::kUInt8) {
      DispatchDequantize<T, uint8_t>(ctx->stream()->As<ep::CudaStream>(), num_bits, symmetric,
                                     outer_size, group_size, inner_size, in->dptr<uint8_t>(),
                                     scale->dptr<T>(), zero == nullptr ? nullptr : zero->dptr<T>(),
                                     out->mut_dptr<T>());
    } else if (in->data_type() == DataType::kInt8) {
      DispatchDequantize<T, int8_t>(ctx->stream()->As<ep::CudaStream>(), num_bits, symmetric,
                                    outer_size, group_size, inner_size, in->dptr<int8_t>(),
                                    scale->dptr<T>(), zero == nullptr ? nullptr : zero->dptr<T>(),
                                    out->mut_dptr<T>());
    } else {
      UNIMPLEMENTED();
    }
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_VECTOR_WISE_SYMMETRIC_DEQUANTIZE_KERNEL(dtype)        \
  REGISTER_USER_KERNEL("groupwise_dequantize")                         \
      .SetCreateFn<GroupwiseDequantizeKernel<dtype>>()                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("scale", 0) == GetDataType<dtype>::value))

REGISTER_VECTOR_WISE_SYMMETRIC_DEQUANTIZE_KERNEL(half);
REGISTER_VECTOR_WISE_SYMMETRIC_DEQUANTIZE_KERNEL(float);

template<typename T, typename C, typename U, int block_size, size_t d_pack_size, size_t q_pack_size,
         int bits, bool symmetric, bool single_group>
__global__ void QuantizedMatmulBiasGroupN(int32_t M, int32_t N, int32_t K, int32_t group_size,
                                          const AlignedArray<T, d_pack_size>* __restrict__ x,
                                          const AlignedArray<U, q_pack_size>* __restrict__ w,
                                          const AlignedArray<T, d_pack_size>* __restrict__ scale,
                                          const AlignedArray<T, d_pack_size>* __restrict__ zero,
                                          const T* __restrict__ bias, T* __restrict__ out) {
  for (int32_t m = blockIdx.x; m < M; m += gridDim.x) {
    const auto* x_m = x + m * K;
    for (int32_t n = blockIdx.y; n < N; n += gridDim.y) {
      C t_sum = 0;
      const auto* w_n = w + n * K;
      const int64_t group_id = single_group ? 0 : n / group_size;
      const auto* scale_n = scale + group_id * K;
      const auto* zero_n = symmetric ? nullptr : zero + group_id * K;
      for (int32_t k = threadIdx.x; k < K; k += block_size) {
        auto xs = x_m[k];
        auto ws = w_n[k];
        auto scale_k = scale_n[k];
        AlignedArray<T, d_pack_size> zero_k;
        if (symmetric) {
          if (std::is_same<U, uint8_t>::value) {
            zero_k = scale_k;
            InplaceMulScalar<T, d_pack_size>()(&zero_k, -static_cast<T>(((1 << (bits - 1)) - 1)));
          } else {
            for (int i = 0; i < d_pack_size; ++i) { zero_k.elem[i] = 0; }
          }
        } else {
          zero_k = zero_n[k];
        }
        AlignedArray<T, d_pack_size> weights;
        Cast<U, T, q_pack_size>()(ws, &weights);
        InplaceFma<T, d_pack_size>()(&weights, scale_k, zero_k);
        MultiplyAccumulate<T, C, d_pack_size>()(xs, weights, &t_sum);
      }
      using BlockReduce = hipcub::BlockReduce<C, block_size>;
      __shared__ typename BlockReduce::TempStorage temp_storage;
      C sum = BlockReduce(temp_storage).Sum(t_sum);
      if (threadIdx.x == 0) {
        if (bias != nullptr) { sum += static_cast<C>(bias[n]); }
        out[m * N + n] = static_cast<T>(sum);
      }
      __syncthreads();
    }
  }
}

template<typename T, typename C, typename U, int num_bits, bool symmetric, size_t d_pack_size,
         size_t q_pack_size, bool single_group>
void LaunchMatmulBiasGroupN(ep::CudaStream* stream, int64_t m, int64_t n, int64_t k,
                            int64_t group_size, const T* x, const U* w, const T* scale,
                            const T* zero, const T* bias, T* out) {
  constexpr uint32_t max_grid_size = 8192;
  constexpr uint32_t block_size = 128;
  const int64_t int32_max = std::numeric_limits<int32_t>::max();
  if (m * k > int32_max || n * k > int32_max || m * n > int32_max || m > int32_max - max_grid_size
      || n > int32_max - max_grid_size || k > int32_max - block_size) {
    UNIMPLEMENTED();
  }
  if constexpr (sizeof(T) * d_pack_size <= 16 && q_pack_size > 0) {
    QuantizedMatmulBiasGroupN<T, C, U, block_size, d_pack_size, q_pack_size, num_bits, symmetric,
                              single_group>
        <<<dim3(std::min<int64_t>(m, max_grid_size), std::min<int64_t>(n, max_grid_size)),
           block_size, 0, stream->hip_stream()>>>(
            m, n, k / d_pack_size, group_size,
            reinterpret_cast<const AlignedArray<T, d_pack_size>*>(x),
            reinterpret_cast<const AlignedArray<U, q_pack_size>*>(w),
            reinterpret_cast<const AlignedArray<T, d_pack_size>*>(scale),
            reinterpret_cast<const AlignedArray<T, d_pack_size>*>(zero), bias, out);
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T, typename C, typename U, int num_bits, bool symmetric, size_t d_pack_size,
         size_t q_pack_size>
void DispatchMatmulBiasGroupNSingleGroup(ep::CudaStream* stream, int64_t m, int64_t n, int64_t k,
                                         int64_t group_size, const T* x, const U* w, const T* scale,
                                         const T* zero, const T* bias, T* out) {
  if (n == group_size) {
    LaunchMatmulBiasGroupN<T, C, U, num_bits, symmetric, d_pack_size, q_pack_size, true>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else {
    LaunchMatmulBiasGroupN<T, C, U, num_bits, symmetric, d_pack_size, q_pack_size, false>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  }
}

template<typename T, typename C, typename U, int num_bits, bool symmetric>
void DispatchMatmulBiasGroupNPackSize(ep::CudaStream* stream, int64_t m, int64_t n, int64_t k,
                                      int64_t group_size, const T* x, const U* w, const T* scale,
                                      const T* zero, const T* bias, T* out) {
  const int max_pack_size = 16 / sizeof(T);
  int pack_size = max_pack_size;
  while (k % pack_size != 0) { pack_size /= 2; }
  constexpr int32_t data_per_quant = 8 / num_bits;
  if (pack_size == 16) {
    DispatchMatmulBiasGroupNSingleGroup<T, C, U, num_bits, symmetric, 16, 16 / data_per_quant>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else if (pack_size == 8) {
    DispatchMatmulBiasGroupNSingleGroup<T, C, U, num_bits, symmetric, 8, 8 / data_per_quant>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else if (pack_size == 4) {
    DispatchMatmulBiasGroupNSingleGroup<T, C, U, num_bits, symmetric, 4, 4 / data_per_quant>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else if (pack_size == 2) {
    DispatchMatmulBiasGroupNSingleGroup<T, C, U, num_bits, symmetric, 2, 2 / data_per_quant>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else if (pack_size == 1) {
    DispatchMatmulBiasGroupNSingleGroup<T, C, U, num_bits, symmetric, 1, 1 / data_per_quant>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T, typename C, typename U, int block_size, size_t d_pack_size, size_t q_pack_size,
         int bits, bool symmetric, bool single_group>
__global__ void QuantizedMatmulBiasGroupK(int32_t M, int32_t N, int32_t K, int32_t group_size,
                                          int32_t num_groups_per_n,
                                          const AlignedArray<T, d_pack_size>* __restrict__ x,
                                          const AlignedArray<U, q_pack_size>* __restrict__ w,
                                          const T* __restrict__ scale, const T* __restrict__ zero,
                                          const T* __restrict__ bias, T* __restrict__ out) {
  for (int32_t m = blockIdx.x; m < M; m += gridDim.x) {
    const auto* x_m = x + m * K;
    for (int32_t n = blockIdx.y; n < N; n += gridDim.y) {
      C t_sum = 0;
      const auto* w_n = w + n * K;
      const auto* scale_n = scale + n * num_groups_per_n;
      const T* zero_n = symmetric ? nullptr : zero + n * num_groups_per_n;
      T group_scale;
      T group_zero;
      if (single_group) {
        group_scale = static_cast<T>(scale_n[0]);
        if (symmetric) {
          if (std::is_same<U, uint8_t>::value) {
            group_zero = -static_cast<T>(((1 << (bits - 1)) - 1)) * group_scale;
          } else {
            group_zero = 0;
          }
        } else {
          group_zero = zero_n[0];
        }
      }
      for (int32_t k = threadIdx.x; k < K; k += block_size) {
        if (!single_group) {
          auto group_id = k / group_size;
          group_scale = static_cast<T>(scale_n[group_id]);
          if (symmetric) {
            if (std::is_same<U, uint8_t>::value) {
              group_zero = -static_cast<T>(((1 << (bits - 1)) - 1)) * group_scale;
            } else {
              group_zero = 0;
            }
          } else {
            group_zero = zero_n[group_id];
          }
        }
        auto xs = x_m[k];
        auto ws = w_n[k];
        AlignedArray<T, d_pack_size> weights;
        Cast<U, T, q_pack_size>()(ws, &weights);
        InplaceFmaScalar<T, d_pack_size>()(&weights, group_scale, group_zero);
        MultiplyAccumulate<T, C, d_pack_size>()(xs, weights, &t_sum);
      }
      using BlockReduce = hipcub::BlockReduce<C, block_size>;
      __shared__ typename BlockReduce::TempStorage temp_storage;
      C sum = BlockReduce(temp_storage).Sum(t_sum);
      if (threadIdx.x == 0) {
        if (bias != nullptr) { sum += static_cast<C>(bias[n]); }
        out[m * N + n] = static_cast<T>(sum);
      }
      __syncthreads();
    }
  }
}

template<typename T, typename C, typename U, int num_bits, bool symmetric, size_t d_pack_size,
         size_t q_pack_size, bool single_group>
void LaunchMatmulBiasGroupK(ep::CudaStream* stream, int64_t m, int64_t n, int64_t k,
                            int64_t group_size, const T* x, const U* w, const T* scale,
                            const T* zero, const T* bias, T* out) {
  constexpr uint32_t max_grid_size = 8192;
  constexpr uint32_t block_size = 128;
  const int64_t int32_max = std::numeric_limits<int32_t>::max();
  if (m * k > int32_max || n * k > int32_max || m * n > int32_max || m > int32_max - max_grid_size
      || n > int32_max - max_grid_size || k > int32_max - block_size) {
    UNIMPLEMENTED();
  }
  if constexpr (sizeof(T) * d_pack_size <= 16 && q_pack_size > 0) {
    QuantizedMatmulBiasGroupK<T, C, U, block_size, d_pack_size, q_pack_size, num_bits, symmetric,
                              single_group>
        <<<dim3(std::min<int64_t>(m, max_grid_size), std::min<int64_t>(n, max_grid_size)),
           block_size, 0, stream->hip_stream()>>>(
            m, n, k / d_pack_size, group_size / d_pack_size, k / group_size,
            reinterpret_cast<const AlignedArray<T, d_pack_size>*>(x),
            reinterpret_cast<const AlignedArray<U, q_pack_size>*>(w), scale, zero, bias, out);
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T, typename C, typename U, int num_bits, bool symmetric, size_t d_pack_size,
         size_t q_pack_size>
void DispatchMatmulBiasGroupKSingleGroup(ep::CudaStream* stream, int64_t m, int64_t n, int64_t k,
                                         int64_t group_size, const T* x, const U* w, const T* scale,
                                         const T* zero, const T* bias, T* out) {
  if (k == group_size) {
    LaunchMatmulBiasGroupK<T, C, U, num_bits, symmetric, d_pack_size, q_pack_size, true>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else {
    LaunchMatmulBiasGroupK<T, C, U, num_bits, symmetric, d_pack_size, q_pack_size, false>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  }
}

template<typename T, typename C, typename U, int num_bits, bool symmetric>
void DispatchMatmulBiasGroupKPackSize(ep::CudaStream* stream, int64_t m, int64_t n, int64_t k,
                                      int64_t group_size, const T* x, const U* w, const T* scale,
                                      const T* zero, const T* bias, T* out) {
  const int max_pack_size = 16 / sizeof(T);
  int pack_size = max_pack_size;
  while (group_size % pack_size != 0) { pack_size /= 2; }
  constexpr int32_t data_per_quant = 8 / num_bits;
  if (pack_size == 16) {
    DispatchMatmulBiasGroupKSingleGroup<T, C, U, num_bits, symmetric, 16, 16 / data_per_quant>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else if (pack_size == 8) {
    DispatchMatmulBiasGroupKSingleGroup<T, C, U, num_bits, symmetric, 8, 8 / data_per_quant>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else if (pack_size == 4) {
    DispatchMatmulBiasGroupKSingleGroup<T, C, U, num_bits, symmetric, 4, 4 / data_per_quant>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else if (pack_size == 2) {
    DispatchMatmulBiasGroupKSingleGroup<T, C, U, num_bits, symmetric, 2, 2 / data_per_quant>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else if (pack_size == 1) {
    DispatchMatmulBiasGroupKSingleGroup<T, C, U, num_bits, symmetric, 1, 1 / data_per_quant>(
        stream, m, n, k, group_size, x, w, scale, zero, bias, out);
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T, typename C, typename U, int num_bits, bool symmetric>
void DispatchMatmulBiasGroupDim(ep::CudaStream* stream, int64_t m, int64_t n, int64_t k,
                                int64_t group_dim, int64_t group_size, const T* x, const U* w,
                                const T* scale, const T* zero, const T* bias, T* out) {
  if (group_dim == 0) {
    DispatchMatmulBiasGroupNPackSize<T, C, U, num_bits, symmetric>(stream, m, n, k, group_size, x,
                                                                   w, scale, zero, bias, out);
  } else if (group_dim == 1) {
    DispatchMatmulBiasGroupKPackSize<T, C, U, num_bits, symmetric>(stream, m, n, k, group_size, x,
                                                                   w, scale, zero, bias, out);
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T, typename C, typename U>
void DispatchMatmulBias(ep::CudaStream* stream, int num_bits, bool symmetric, int64_t m, int64_t n,
                        int64_t k, int64_t group_dim, int64_t group_size, const T* x, const U* w,
                        const T* scale, const T* zero, const T* bias, T* out) {
  if (num_bits == 4) {
    if (symmetric) {
      DispatchMatmulBiasGroupDim<T, C, U, 4, true>(stream, m, n, k, group_dim, group_size, x, w,
                                                   scale, zero, bias, out);
    } else {
      DispatchMatmulBiasGroupDim<T, C, U, 4, false>(stream, m, n, k, group_dim, group_size, x, w,
                                                    scale, zero, bias, out);
    }
  } else if (num_bits == 8) {
    if (symmetric) {
      DispatchMatmulBiasGroupDim<T, C, U, 8, true>(stream, m, n, k, group_dim, group_size, x, w,
                                                   scale, zero, bias, out);
    } else {
      DispatchMatmulBiasGroupDim<T, C, U, 8, false>(stream, m, n, k, group_dim, group_size, x, w,
                                                    scale, zero, bias, out);
    }
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T>
class FusedLinearWithGroupwiseQuantizedWeightKernel final : public user_op::OpKernel,
                                                            public user_op::CudaGraphSupport {
 public:
  FusedLinearWithGroupwiseQuantizedWeightKernel() = default;
  ~FusedLinearWithGroupwiseQuantizedWeightKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState*,
               const user_op::OpKernelCache* cache) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* w = ctx->Tensor4ArgNameAndIndex("w", 0);
    const user_op::Tensor* w_scale = ctx->Tensor4ArgNameAndIndex("w_scale", 0);
    const user_op::Tensor* b =
        (ctx->has_input("b", 0)) ? ctx->Tensor4ArgNameAndIndex("b", 0) : nullptr;
    const user_op::Tensor* w_zero =
        (ctx->has_input("w_zero", 0)) ? ctx->Tensor4ArgNameAndIndex("w_zero", 0) : nullptr;
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const DataType data_type = x->data_type();
    CHECK_EQ(w_scale->data_type(), data_type);
    CHECK_EQ(out->data_type(), data_type);
    const int64_t group_size = ctx->Attr<int64_t>("group_size");
    const int64_t group_dim = ctx->Attr<int64_t>("group_dim");
    CHECK(group_dim == 0 || group_dim == 1);
    const int32_t num_bits = ctx->Attr<int32_t>("num_bits");
    const bool symmetric = ctx->Attr<bool>("symmetric");
    CHECK_GE(x->shape_view().NumAxes(), 2);
    const int64_t k = x->shape_view().At(x->shape_view().NumAxes() - 1);
    const int64_t m = x->shape_view().elem_cnt() / k;
    CHECK_EQ(w->shape_view().NumAxes(), 2);
    if (num_bits == 4) {
      CHECK_EQ(w->shape_view().At(1) * 2, k);
    } else if (num_bits == 8) {
      CHECK_EQ(w->shape_view().At(1), k);
    } else {
      UNIMPLEMENTED();
    }
    const int64_t n = w->shape_view().At(0);
    const int64_t group_dim_size = group_dim == 0 ? n : k;
    CHECK_GT(group_size, 0);
    CHECK_LE(group_size, group_dim_size);
    CHECK_EQ(group_dim_size % group_size, 0);
    const int64_t num_groups = group_dim_size / group_size;
    if (group_dim == 0) {
      CHECK_EQ(w_scale->shape_view().At(0), num_groups);
      CHECK_EQ(w_scale->shape_view().At(1), k);
    } else if (group_dim == 1) {
      CHECK_EQ(w_scale->shape_view().At(0), n);
      CHECK_EQ(w_scale->shape_view().At(1), num_groups);
    } else {
      UNIMPLEMENTED();
    }
    if (w_zero != nullptr) {
      CHECK_EQ(w_zero->data_type(), data_type);
      CHECK(w_zero->shape_view() == w_scale->shape_view());
    }
    if (b != nullptr) {
      CHECK_EQ(b->data_type(), data_type);
      CHECK_EQ(b->shape_view().NumAxes(), 1);
      CHECK_EQ(b->shape_view().At(0), n);
    }
    CHECK_EQ(x->shape_view().NumAxes(), out->shape_view().NumAxes());
    for (int i = 0; i < x->shape_view().NumAxes() - 1; ++i) {
      CHECK_EQ(out->shape_view().At(i), x->shape_view().At(i));
    }
    CHECK_EQ(out->shape_view().At(out->shape_view().NumAxes() - 1), n);
    if (symmetric) {
      CHECK(w_zero == nullptr);
    } else {
      CHECK(w_zero != nullptr);
    }
    const DataType quant_type = w->data_type();
    if (quant_type == DataType::kUInt8) {
      DispatchMatmulBias<T, float, uint8_t>(
          ctx->stream()->As<ep::CudaStream>(), num_bits, symmetric, m, n, k, group_dim, group_size,
          x->dptr<T>(), w->dptr<uint8_t>(), w_scale->dptr<T>(),
          w_zero == nullptr ? nullptr : w_zero->dptr<T>(), b == nullptr ? nullptr : b->dptr<T>(),
          out->mut_dptr<T>());
    } else if (quant_type == DataType::kInt8) {
      DispatchMatmulBias<T, float, int8_t>(
          ctx->stream()->As<ep::CudaStream>(), num_bits, symmetric, m, n, k, group_dim, group_size,
          x->dptr<T>(), w->dptr<int8_t>(), w_scale->dptr<T>(),
          w_zero == nullptr ? nullptr : w_zero->dptr<T>(), b == nullptr ? nullptr : b->dptr<T>(),
          out->mut_dptr<T>());
    } else {
      UNIMPLEMENTED();
    }
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_MATMUL_BIAS_KERNEL_GPU(data_type, cpp_type)            \
  REGISTER_USER_KERNEL("fused_linear_with_groupwise_quantized_weight")        \
      .SetCreateFn<FusedLinearWithGroupwiseQuantizedWeightKernel<cpp_type>>() \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)        \
                       && (user_op::HobDataType("out", 0) == data_type));

REGISTER_FUSED_MATMUL_BIAS_KERNEL_GPU(DataType::kFloat, float);
REGISTER_FUSED_MATMUL_BIAS_KERNEL_GPU(DataType::kFloat16, half);

}  // namespace

}  // namespace oneflow
