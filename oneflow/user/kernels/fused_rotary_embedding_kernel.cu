#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

void ParseDims(const ShapeView& shape, const std::string& x_layout,
               const Optional<int64_t>& num_heads, const Optional<int64_t>& head_size,
               int64_t tensor_index, int64_t* b, int64_t* m, int64_t* h, int64_t* k,
               int64_t* b_stride, int64_t* m_stride, int64_t* h_stride, int64_t* offset) {
  if (shape.NumAxes() == 3) {
    if (x_layout == "BM(HK)" || x_layout == "BM(H2K)" || x_layout == "BM(H3K)"
        || x_layout == "MB(HK)" || x_layout == "MB(H2K)" || x_layout == "MB(H3K)") {
      bool batch_first = false;
      int64_t packed_n = 0;
      const std::string x_layout_bm = x_layout.substr(0, 2);
      const std::string x_layout_hk = x_layout.substr(2);
      if (x_layout_bm == "BM") {
        *b = shape.At(0);
        *m = shape.At(1);
        batch_first = true;
      } else if (x_layout_bm == "MB") {
        *b = shape.At(1);
        *m = shape.At(0);
        batch_first = false;
      } else {
        UNIMPLEMENTED();
      }
      if (x_layout_hk == "(HK)") {
        packed_n = 1;
      } else if (x_layout_hk == "(H2K)") {
        packed_n = 2;
      } else if (x_layout_hk == "(H3K)") {
        packed_n = 3;
      } else {
        UNIMPLEMENTED();
      }
      const int64_t hidden_size = shape.At(2);
      if (num_heads) {
        const int64_t expected_h = CHECK_JUST(num_heads);
        const int64_t packed_h = packed_n * expected_h;
        CHECK_EQ(hidden_size % packed_h, 0);
        *h = expected_h;
        *k = hidden_size / packed_h;
      } else if (head_size) {
        const int64_t expected_k = CHECK_JUST(head_size);
        const int64_t packed_k = packed_n * expected_k;
        CHECK_EQ(hidden_size % packed_k, 0);
        *h = hidden_size / packed_k;
        *k = expected_k;
      } else {
        UNIMPLEMENTED();
      }
      *h_stride = *k * packed_n;
      if (batch_first) {
        *m_stride = *h_stride * *h;
        *b_stride = *m_stride * *m;
      } else {
        *b_stride = *h_stride * *h;
        *m_stride = *b_stride * *b;
      }
      if (packed_n == 1) {
        *offset = 0;
      } else if (packed_n == 2) {
        CHECK_GE(tensor_index, 1);
        *offset = (tensor_index - 1) * *k;
      } else if (packed_n == 3) {
        *offset = tensor_index * *k;
      } else {
        UNIMPLEMENTED();
      }
    } else {
      UNIMPLEMENTED();
    }
  } else if (shape.NumAxes() == 4) {
    if (x_layout == "BMHK") {
      *b = shape.At(0);
      *m = shape.At(1);
      *h = shape.At(2);
      *k = shape.At(3);
      *h_stride = *k;
      *m_stride = *h_stride * *h;
      *b_stride = *m_stride * *m;
    } else if (x_layout == "BHMK") {
      *b = shape.At(0);
      *m = shape.At(2);
      *h = shape.At(1);
      *k = shape.At(3);
      *m_stride = *k;
      *h_stride = *m_stride * *m;
      *b_stride = *h_stride * *h;
    } else if (x_layout == "MBHK") {
      *b = shape.At(1);
      *m = shape.At(0);
      *h = shape.At(2);
      *k = shape.At(3);
      *h_stride = *k;
      *b_stride = *h_stride * *h;
      *m_stride = *b_stride * *b;
    } else {
      UNIMPLEMENTED();
    }
    if (num_heads) {
      const int64_t expected_h = CHECK_JUST(num_heads);
      CHECK_EQ(*h, expected_h);
    }
    if (head_size) {
      const int64_t expected_k = CHECK_JUST(head_size);
      CHECK_EQ(*k, expected_k);
    }
    *offset = 0;
  } else {
    UNIMPLEMENTED();
  };
}

template<typename T, typename PositionType, typename IndexType, int num_dims>
struct FusedApplyRotaryEmbParam {
  const T* x;
  const T* cos;
  const T* sin;
  const PositionType* position_ids;
  T* out;
  T theta;
  int64_t pass_ndims;
  int64_t rotary_emb_dim;  // TODO: dispatch
  IndexType k0;
  IndexType k1;
  IndexType num_elements;
  IndexType k;
  IndexType sinuous_m;
  IndexType offset;
  IndexType x_stride[num_dims];
  IndexType position_stride[num_dims];
  IndexType position_mask[num_dims];
  IndexType sinuous_stride[num_dims];
  IndexType sinuous_mask[num_dims];

  // TODO: position_id type? for now it is int32
  FusedApplyRotaryEmbParam(const T* x, const T* cos, const T* sin, const PositionType* position_ids,
                           T* out, const T theta, const int64_t pass_ndims,
                           const int rotary_emb_dim, const IndexType num_elements,
                           const IndexType k, const IndexType k0, const IndexType k1,
                           const IndexType sinuous_m, const IndexType offset)
      : x(x),
        cos(cos),
        sin(sin),
        position_ids(position_ids),
        out(out),
        theta(theta),
        pass_ndims(pass_ndims),
        rotary_emb_dim(rotary_emb_dim),
        num_elements(num_elements),
        k(k),
        k0(k0),
        k1(k1),
        sinuous_m(sinuous_m),
        offset(offset) {}
};

template<typename T, typename PositionType, typename IndexType, size_t pack_size, size_t num_dims>
__global__ void FusedApplyRotaryEmbFetchKernel(
    FusedApplyRotaryEmbParam<T, PositionType, IndexType, num_dims> param) {
  const T* x = param.x;
  const T* cos = param.cos;
  const T* sin = param.sin;
  const PositionType* position_ids = param.position_ids;
  T* out = param.out;
  const IndexType packed_k = param.k / pack_size;
  const IndexType packed_pass_ndims = param.pass_ndims / pack_size;
  const IndexType packed_rotary_ndims = packed_k - packed_pass_ndims;
  const IndexType rotary_num_elements = param.num_elements / packed_k * packed_rotary_ndims;
  for (IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x;
       packed_offset < rotary_num_elements; packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset =
        param.offset
        + (packed_offset % packed_rotary_ndims + packed_offset / packed_rotary_ndims * packed_k)
              * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;
    IndexType sinuous_offset = 0;
    IndexType position_id_offset = 0;
    IndexType k_index = 0;

    IndexType temp_offset = offset;
#pragma unloop
    for (int i = 0; i < num_dims; i++) {
      IndexType index = temp_offset / param.x_stride[i];
      if (i == num_dims - 1) k_index = index;
      temp_offset = temp_offset - index * param.x_stride[i];
      position_id_offset =
          position_id_offset + (index * param.position_mask[i] * param.position_stride[i]);
    }

    position_id_offset =
        position_id_offset
        + param.sinuous_m * (k_index * param.rotary_emb_dim / (param.k - param.pass_ndims));

    const int position = position_ids[position_id_offset];

    sinuous_offset = position * param.k + k_index;

    const LoadPack* cos_load = reinterpret_cast<const LoadPack*>(cos + sinuous_offset);
    const LoadPack* sin_load = reinterpret_cast<const LoadPack*>(sin + sinuous_offset);
    const LoadPack cos_vec = *cos_load, sin_vec = *sin_load;
    LoadPack out_vec;

#pragma unloop
    for (int i = 0; i < pack_size / 2; i++) {
      out_vec.elem[i * 2] =
          x_vec.elem[i * 2] * cos_vec.elem[i * 2] - x_vec.elem[i * 2 + 1] * sin_vec.elem[i * 2];
      out_vec.elem[i * 2 + 1] = x_vec.elem[i * 2 + 1] * cos_vec.elem[i * 2 + 1]
                                + x_vec.elem[i * 2] * sin_vec.elem[i * 2 + 1];
    }

    *(reinterpret_cast<LoadPack*>(out + offset)) = out_vec;
  }

  for (IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x;
       packed_offset < param.num_elements && packed_offset >= rotary_num_elements;
       packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset =
        param.offset
        + (packed_rotary_ndims + (packed_offset - rotary_num_elements) % packed_pass_ndims
           + (packed_offset - rotary_num_elements) / packed_pass_ndims * packed_k)
              * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;

    *(reinterpret_cast<LoadPack*>(out + offset)) = x_vec;
  }
}

template<typename T, typename PositionType, typename IndexType, size_t pack_size, size_t num_dims>
__global__ void FusedApplyRotaryEmbFetchWithoutPositionKernel(
    FusedApplyRotaryEmbParam<T, PositionType, IndexType, num_dims> param) {
  const T* x = param.x;
  const T* cos = param.cos;
  const T* sin = param.sin;
  T* out = param.out;
  const IndexType packed_k = param.k / pack_size;
  const IndexType packed_pass_ndims = param.pass_ndims / pack_size;
  const IndexType packed_rotary_ndims = packed_k - packed_pass_ndims;
  const IndexType rotary_num_elements = param.num_elements / packed_k * packed_rotary_ndims;
  for (IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x;
       packed_offset < rotary_num_elements; packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset =
        param.offset
        + (packed_offset % packed_rotary_ndims + packed_offset / packed_rotary_ndims * packed_k)
              * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;
    IndexType sinuous_offset = 0;
    IndexType k_index = 0;

    IndexType temp_offset = offset;
#pragma unloop
    for (int i = 0; i < num_dims; i++) {
      IndexType index = temp_offset / param.x_stride[i];
      temp_offset = temp_offset - index * param.x_stride[i];
      sinuous_offset = sinuous_offset + index * param.sinuous_stride[i] * param.sinuous_mask[i];
    }

    const LoadPack* cos_load = reinterpret_cast<const LoadPack*>(cos + sinuous_offset);
    const LoadPack* sin_load = reinterpret_cast<const LoadPack*>(sin + sinuous_offset);
    const LoadPack cos_vec = *cos_load, sin_vec = *sin_load;
    LoadPack out_vec;

#pragma unloop
    for (int i = 0; i < pack_size / 2; i++) {
      out_vec.elem[i * 2] =
          x_vec.elem[i * 2] * cos_vec.elem[i * 2] - x_vec.elem[i * 2 + 1] * sin_vec.elem[i * 2];
      out_vec.elem[i * 2 + 1] = x_vec.elem[i * 2 + 1] * cos_vec.elem[i * 2 + 1]
                                + x_vec.elem[i * 2] * sin_vec.elem[i * 2 + 1];
    }

    *(reinterpret_cast<LoadPack*>(out + offset)) = out_vec;
  }

  for (IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x;
       packed_offset < param.num_elements && packed_offset >= rotary_num_elements;
       packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset =
        param.offset
        + (packed_rotary_ndims + (packed_offset - rotary_num_elements) % packed_pass_ndims
           + (packed_offset - rotary_num_elements) / packed_pass_ndims * packed_k)
              * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;

    *(reinterpret_cast<LoadPack*>(out + offset)) = x_vec;
  }
}

template<typename T, typename PositionType, typename IndexType, size_t pack_size, size_t num_dims>
__global__ void FusedApplyRotaryEmbComputeWithoutPositionKernel(
    FusedApplyRotaryEmbParam<T, PositionType, IndexType, num_dims> param) {
  const T* x = param.x;
  const T* cos = param.cos;
  const T* sin = param.sin;
  T* out = param.out;
  const T theta = param.theta;
  const IndexType packed_k = param.k / pack_size;
  const IndexType packed_pass_ndims = param.pass_ndims / pack_size;
  const IndexType packed_rotary_ndims = packed_k - packed_pass_ndims;
  const IndexType rotary_num_elements = param.num_elements / packed_k * packed_rotary_ndims;
  for (IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x;
       packed_offset < rotary_num_elements; packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset =
        param.offset
        + (packed_offset % packed_rotary_ndims + packed_offset / packed_rotary_ndims * packed_k)
              * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;
    IndexType sinuous_offset = 0;
    IndexType k_index = 0;

    IndexType temp_offset = offset;
#pragma unloop
    for (int i = 0; i < num_dims; i++) {
      IndexType index = temp_offset / param.x_stride[i];
      if (i == num_dims - 1) k_index = index;
      temp_offset = temp_offset - index * param.x_stride[i];
      sinuous_offset = sinuous_offset + index * param.sinuous_stride[i] * param.sinuous_mask[i];
    }

    IndexType m_index = sinuous_offset / param.k;

    LoadPack cos_vec, sin_vec, out_vec;

#pragma unloop
    for (int i = 0; i < pack_size / 2; i++) {
      float val =
          m_index * expf(2.0f * static_cast<float>(k_index / 2 + i) / param.k * logf(theta));
      T cos_val = cosf(val);
      T sin_val = sinf(val);
      cos_vec.elem[i * 2] = cos_val;
      cos_vec.elem[i * 2 + 1] = cos_val;
      sin_vec.elem[i * 2] = sin_val;
      sin_vec.elem[i * 2 + 1] = sin_val;
    }

#pragma unloop
    for (int i = 0; i < pack_size / 2; i++) {
      out_vec.elem[i * 2] =
          x_vec.elem[i * 2] * cos_vec.elem[i * 2] - x_vec.elem[i * 2 + 1] * sin_vec.elem[i * 2];
      out_vec.elem[i * 2 + 1] = x_vec.elem[i * 2 + 1] * cos_vec.elem[i * 2 + 1]
                                + x_vec.elem[i * 2] * sin_vec.elem[i * 2 + 1];
    }

    *(reinterpret_cast<LoadPack*>(out + offset)) = out_vec;
  }

  for (IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x;
       packed_offset < param.num_elements && packed_offset >= rotary_num_elements;
       packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset =
        param.offset
        + (packed_rotary_ndims + (packed_offset - rotary_num_elements) % packed_pass_ndims
           + (packed_offset - rotary_num_elements) / packed_pass_ndims * packed_k)
              * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;

    *(reinterpret_cast<LoadPack*>(out + offset)) = x_vec;
  }
}

template<typename T, typename PositionType, typename IndexType, size_t pack_size, size_t num_dims>
__global__ void FusedApplyRotaryEmbComputeKernel(
    FusedApplyRotaryEmbParam<T, PositionType, IndexType, num_dims> param) {
  const T* x = param.x;
  const T* cos = param.cos;
  const T* sin = param.sin;
  const PositionType* position_ids = param.position_ids;
  T* out = param.out;
  const T theta = param.theta;
  const IndexType packed_k = param.k / pack_size;
  const IndexType packed_pass_ndims = param.pass_ndims / pack_size;
  const IndexType packed_rotary_ndims = packed_k - packed_pass_ndims;
  const IndexType rotary_num_elements = param.num_elements / packed_k * packed_rotary_ndims;
  for (IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x;
       packed_offset < rotary_num_elements; packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset =
        param.offset
        + (packed_offset % packed_rotary_ndims + packed_offset / packed_rotary_ndims * packed_k)
              * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;
    IndexType position_id_offset = 0;
    IndexType k_index = 0;

    IndexType temp_offset = offset;
#pragma unloop
    for (int i = 0; i < num_dims; i++) {
      IndexType index = temp_offset / param.x_stride[i];
      if (i == num_dims - 1) k_index = index;
      temp_offset = temp_offset - index * param.x_stride[i];
      position_id_offset =
          position_id_offset + (index * param.position_mask[i] * param.position_stride[i]);
    }

    position_id_offset =
        position_id_offset
        + param.sinuous_m * (k_index * param.rotary_emb_dim / (param.k - param.pass_ndims));

    const PositionType position = position_ids[position_id_offset];

    LoadPack cos_vec, sin_vec, out_vec;

#pragma unloop
    for (int i = 0; i < pack_size / 2; i++) {
      float val =
          position * expf(2.0f * static_cast<float>(k_index / 2 + i) / param.k * logf(theta));
      T cos_val = cosf(val);
      T sin_val = sinf(val);
      cos_vec.elem[i * 2] = cos_val;
      cos_vec.elem[i * 2 + 1] = cos_val;
      sin_vec.elem[i * 2] = sin_val;
      sin_vec.elem[i * 2 + 1] = sin_val;
    }

#pragma unloop
    for (int i = 0; i < pack_size / 2; i++) {
      out_vec.elem[i * 2] =
          x_vec.elem[i * 2] * cos_vec.elem[i * 2] - x_vec.elem[i * 2 + 1] * sin_vec.elem[i * 2];
      out_vec.elem[i * 2 + 1] = x_vec.elem[i * 2 + 1] * cos_vec.elem[i * 2 + 1]
                                + x_vec.elem[i * 2] * sin_vec.elem[i * 2 + 1];
    }

    *(reinterpret_cast<LoadPack*>(out + offset)) = out_vec;
  }

  for (IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x;
       packed_offset < param.num_elements && packed_offset >= rotary_num_elements;
       packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset =
        param.offset
        + (packed_rotary_ndims + (packed_offset - rotary_num_elements) % packed_pass_ndims
           + (packed_offset - rotary_num_elements) / packed_pass_ndims * packed_k)
              * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;

    *(reinterpret_cast<LoadPack*>(out + offset)) = x_vec;
  }
}

template<typename T, typename PositionType, typename IndexType, size_t num_dims>
__global__ void PlaneKernel(FusedApplyRotaryEmbParam<T, PositionType, IndexType, num_dims> param) {
  const T* x = param.x;
  const T* cos = param.cos;
  const T* sin = param.sin;
  const PositionType* position_ids = param.position_ids;
  T* out = param.out;
  IndexType sinuous_offset = 0;
  const T theta = param.theta;

  for (IndexType offset = threadIdx.x + blockIdx.x * blockDim.x; offset < param.num_elements;
       offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, 2>;
    IndexType temp_offset = offset;
    IndexType k_index, m_index;
    IndexType position_id_offset = 0;
    PositionType position;
#pragma unloop
    for (int i = 0; i < num_dims; i++) {
      IndexType index = temp_offset / param.x_stride[i];
      if (i == num_dims - 1) k_index = index;
      temp_offset = temp_offset - index * param.x_stride[i];
      sinuous_offset = sinuous_offset + index * param.sinuous_stride[i] * param.sinuous_mask[i];
      position_id_offset =
          position_id_offset + (index * param.position_mask[i] * param.position_stride[i]);
    }

    m_index = sinuous_offset / param.k;
    position_id_offset =
        position_id_offset
        + param.sinuous_m * (k_index * param.rotary_emb_dim / (param.k - param.pass_ndims));

    if (param.position_ids) {
      position = position_ids[position_id_offset];
      sinuous_offset = position * param.k + k_index;
    } else {
      position = m_index;
    }

    LoadPack x_vec;

    T cos_val;
    T sin_val;
    T out_val;

    if (param.cos) {
      cos_val = *(cos + sinuous_offset);
      sin_val = *(sin + sinuous_offset);
    } else {
      int actual_ndim = (param.k - param.pass_ndims) / param.rotary_emb_dim;
      float val = position
                  * expf(2.0f * static_cast<float>(k_index % (actual_ndim / 2)) / actual_ndim
                         * logf(theta));
      cos_val = cosf(val);
      sin_val = sinf(val);
    }

    if (k_index < param.k0) {
      x_vec.elem[0] = *(x + offset);
      x_vec.elem[1] = (k_index < param.k0 / 2) ? static_cast<T>(-*(x + offset + param.k0 / 2))
                                               : *(x + offset - param.k0 / 2);
      out_val = cos_val * x_vec.elem[0] + sin_val * x_vec.elem[1];
    } else if (k_index < param.k1) {
      if ((k_index - param.k0) < ((param.k - param.pass_ndims) / (2 * param.rotary_emb_dim))) {
        x_vec.elem[0] = *(x + offset);
        x_vec.elem[1] = -*(x + offset + param.k0 / 2);
        out_val = cos_val * x_vec.elem[0] + sin_val * x_vec.elem[1];
      } else {
        x_vec.elem[0] = *(x + offset);
        x_vec.elem[1] = *(x + offset - param.k0 / 2);
        out_val = cos_val * x_vec.elem[0] + sin_val * x_vec.elem[1];
      }
    } else {
      *(out + offset) = *(x + offset);
    }

    *(out + offset) = out_val;
  }
}

template<typename T, typename PositionType, typename IndexType, size_t pack_size, size_t num_dims>
void LaunchKernel(ep::CudaStream* stream, const T* x, const T* cos, const T* sin, const PositionType* position_ids, T* out,
                  const int64_t* position_shape, const std::string& x_layout,
                  const std::string& output_layout, const std::string& mode, const T theta,
                  const int64_t pass_ndims, const int rotary_emb_dim, const int64_t b,
                  const int64_t m, const int64_t h, const int64_t k, const int64_t b_stride,
                  const int64_t m_stride, const int64_t h_stride, const int64_t offset,
                  IndexType num_elements) {
  DimVector kernel_x_shape(num_dims), kernel_sinuous_shape(num_dims);
  size_t x_stride[num_dims];
  size_t sinuous_stride[num_dims];

  x_stride[num_dims - 1] = 1;
  sinuous_stride[num_dims - 1] = 1;

  for (int i = num_dims - 2; i >= 0; i--) {
    x_stride[i] = x_stride[i + 1] * kernel_x_shape.at(i + 1);
    sinuous_stride[i] = sinuous_stride[i + 1] * kernel_sinuous_shape.at(i + 1);
  }

  IndexType k0, k1;

  if (rotary_emb_dim == 1) {
    k0 = k - pass_ndims;
    k1 = k - pass_ndims;
  } else {
    k0 = (k - pass_ndims) / 2;
    k1 = k - pass_ndims;
  }

  struct FusedApplyRotaryEmbParam<T, PositionType, IndexType, num_dims> param(
      x, cos, sin, position_ids, out, theta, pass_ndims, rotary_emb_dim, num_elements, k, k0, k1,
      position_shape ? static_cast<IndexType>(position_shape[2]) : m, offset);

  std::pair<char, std::int64_t> strides[num_dims];
  strides[0] = {'b', b_stride};
  strides[1] = {'h', h_stride};
  strides[2] = {'m', m_stride};
  strides[3] = {'k', 1};

  auto GetDimX = [&](const char c) {
    if (c == 'b') {
      return b;
    } else if (c == 'h') {
      return h;
    } else if (c == 'm') {
      return m;
    } else if (c == 'k') {
      return k;
    }

    return 0L;
  };

  std::sort(strides, strides + num_dims, [&](auto pair1, auto pair2) {
    if (pair1.second > pair2.second) {
      return true;
    } else if (pair1.second == pair2.second) {
      if (GetDimX(pair1.first) != 1) { return true; }
      return false;
    } else {
      return false;
    }
    return pair1.second > pair2.second;
  });

// K has to be the last dimension, only k&m matters, therefore strides other than k&m does not
// really needs to be computed
#pragma unloop
  for (int i = 0; i < num_dims; i++) {
    param.x_stride[i] = strides[i].second;
    param.sinuous_mask[i] = 0;
    param.position_mask[i] = 0;
    if (strides[i].first == 'm') {
      param.sinuous_mask[i] = 1;
      param.sinuous_stride[i] = k;
      param.position_stride[i] = 1;
      param.position_mask[i] = 1;
    } else if (strides[i].first == 'k') {
      param.sinuous_mask[i] = 1;
      param.sinuous_stride[i] = 1;
    } else if (strides[i].first == 'b') {
      param.position_stride[i] = rotary_emb_dim * param.sinuous_m;
      param.position_mask[i] = 1;
    }
  }

  constexpr size_t blk_size = 128;

  // TODO: something bad here... need to be refined to support mode dispatch
  if (mode == "plane") {
    param.num_elements = param.num_elements * pack_size;
    PlaneKernel<T, PositionType, IndexType, num_dims>
        <<<(param.num_elements + blk_size - 1) / blk_size, blk_size, 0, stream->hip_stream()>>>(param);
    return;
  }

  if (cos) {
    if (position_ids) {
      FusedApplyRotaryEmbFetchKernel<T, PositionType, IndexType, pack_size, num_dims>
          <<<(param.num_elements + blk_size - 1) / blk_size, blk_size, 0, stream->hip_stream()>>>(param);
    } else {
      FusedApplyRotaryEmbFetchWithoutPositionKernel<T, PositionType, IndexType, pack_size, num_dims>
          <<<(param.num_elements + blk_size - 1) / blk_size, blk_size, 0, stream->hip_stream()>>>(param);
    }
  } else {
    if (position_ids) {
      FusedApplyRotaryEmbComputeKernel<T, PositionType, IndexType, pack_size, num_dims>
          <<<(param.num_elements + blk_size - 1) / blk_size, blk_size, 0, stream->hip_stream()>>>(param);
    } else {
      FusedApplyRotaryEmbComputeWithoutPositionKernel<T, PositionType, IndexType, pack_size,
                                                      num_dims>
          <<<(param.num_elements + blk_size - 1) / blk_size, blk_size, 0, stream->hip_stream()>>>(param);
    }
  }
}

template<typename T, typename PositionType, typename IndexType, size_t num_dims>
void DispatchPackSize(ep::CudaStream* stream, const T* x, const T* cos, const T* sin, const PositionType* position_ids,
                      T* out, const int64_t* position_shape, const std::string& x_layout,
                      const std::string& output_layout, const std::string& mode, const T theta,
                      const int64_t pass_ndims, const int rotary_emb_dim, const IndexType b,
                      const IndexType m, const IndexType h, const IndexType k,
                      const IndexType b_stride, const IndexType m_stride, const IndexType h_stride,
                      const IndexType offset, IndexType num_elements) {
  const auto CheckPackSize = [&](const size_t pack_size) {
    bool r = (((reinterpret_cast<uintptr_t>(x) % (sizeof(T) * pack_size)) == 0)
              && ((((k - pass_ndims) / 2) % pack_size) == 0) && ((pass_ndims % pack_size) == 0)
              && ((16 / sizeof(T)) >= pack_size));
    return r;
  };

  if (CheckPackSize(8)) {
    num_elements /= 8;
    LaunchKernel<T, PositionType, IndexType, 8, num_dims>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode, theta,
        pass_ndims, rotary_emb_dim, b, m, h, k, b_stride, m_stride, h_stride, offset, num_elements);
  } else if (CheckPackSize(4)) {
    num_elements /= 4;
    LaunchKernel<T, PositionType, IndexType, 4, num_dims>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode, theta,
        pass_ndims, rotary_emb_dim, b, m, h, k, b_stride, m_stride, h_stride, offset, num_elements);
  } else {
    num_elements /= 2;
    LaunchKernel<T, PositionType, IndexType, 2, num_dims>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode, theta,
        pass_ndims, rotary_emb_dim, b, m, h, k, b_stride, m_stride, h_stride, offset, num_elements);
  }
}

template<typename T, typename PositionType, size_t num_dims>
void DispatchIndex(ep::CudaStream* stream, const T* x, const T* cos, const T* sin, const PositionType* position_ids, T* out,
                   const int64_t* position_shape, const std::string& x_layout,
                   const std::string& output_layout, const std::string& mode, const T theta,
                   const int64_t pass_ndims, const int rotary_emb_dim, const int64_t b,
                   const int64_t m, const int64_t h, const int64_t k, const int64_t b_stride,
                   const int64_t m_stride, const int64_t h_stride, const int64_t offset) {
  int64_t num_elements = b * m * h * k;
  if (num_elements < (1 << 30)) {
    DispatchPackSize<T, PositionType, int32_t, num_dims>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode, theta,
        pass_ndims, rotary_emb_dim, static_cast<int32_t>(b), static_cast<int32_t>(m),
        static_cast<int32_t>(h), static_cast<int32_t>(k), static_cast<int32_t>(b_stride),
        static_cast<int32_t>(m_stride), static_cast<int32_t>(h_stride),
        static_cast<int32_t>(offset), static_cast<int32_t>(num_elements));
  } else {
    DispatchPackSize<T, PositionType, int64_t, num_dims>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode, theta,
        pass_ndims, rotary_emb_dim, b, m, h, k, b_stride, m_stride, h_stride, offset, num_elements);
  }
}

template<typename T, typename PositionType>
class FusedApplyRotaryEmbKernel final : public user_op::OpKernel {
 public:
  FusedApplyRotaryEmbKernel() = default;
  ~FusedApplyRotaryEmbKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* cos = nullptr;
    user_op::Tensor* sin = nullptr;
    user_op::Tensor* position_ids = nullptr;
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const std::string& x_layout = ctx->Attr<std::string>("x_layout");
    const std::string& output_layout = ctx->Attr<std::string>("output_layout");
    const std::string& mode = ctx->Attr<std::string>("mode");
    const int64_t tensor_index = ctx->Attr<int64_t>("tensor_index");
    const int64_t k_size = ctx->Attr<int64_t>("k_size");
    const int64_t pass_ndims = k_size - ctx->Attr<int64_t>("rotary_size");
    const float theta = 1.0f / ctx->Attr<float>("base");
    int rotary_emb_dim = 1;

    if (ctx->has_input("cos", 0)) { cos = ctx->Tensor4ArgNameAndIndex("cos", 0); }

    if (ctx->has_input("sin", 0)) { sin = ctx->Tensor4ArgNameAndIndex("sin", 0); }

    if (ctx->has_input("position_ids", 0)) {
      position_ids = ctx->Tensor4ArgNameAndIndex("position_ids", 0);
      rotary_emb_dim = position_ids->shape_view().At(1);
    }

    constexpr size_t ndims = 4;
    int64_t b = 0;
    int64_t m = 0;
    int64_t h = 0;
    int64_t k = 0;
    int64_t b_stride = 0;
    int64_t m_stride = 0;
    int64_t h_stride = 0;
    int64_t offset = 0;

    ParseDims(x->shape_view(), x_layout, Optional<int64_t>(),
              k_size ? Optional<int64_t>(k_size) : Optional<int64_t>(), tensor_index, &b, &m, &h,
              &k, &b_stride, &m_stride, &h_stride, &offset);

    // TODO: hard code num_dims & seems redundant template problem...
    DispatchIndex<T, PositionType, ndims>(
        ctx->stream()->As<ep::CudaStream>(),
        reinterpret_cast<const T*>(x->dptr()),
        cos ? reinterpret_cast<const T*>(cos->dptr()) : nullptr,
        sin ? reinterpret_cast<const T*>(sin->dptr()) : nullptr,
        position_ids ? reinterpret_cast<const PositionType*>(position_ids->dptr()) : nullptr,
        reinterpret_cast<T*>(out->mut_dptr()),
        position_ids ? position_ids->shape_view().data() : nullptr, x_layout, output_layout, mode,
        static_cast<T>(theta), pass_ndims, rotary_emb_dim, b, m, h, k, b_stride, m_stride, h_stride,
        offset);
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_APPLY_ROTARY_EMB_GPU(dtype, position_type)          \
  REGISTER_USER_KERNEL("fused_apply_rotary_emb")                           \
      .SetCreateFn<FusedApplyRotaryEmbKernel<dtype, position_type>>()      \
      .SetIsMatchedHob(                                                    \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                  \
          && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value) \
          && (user_op::HobInputSize("position_ids") == 1)                  \
          && (user_op::HobDataType("position_ids", 0) == GetDataType<position_type>::value));

#define REGISTER_FUSED_APPLY_ROTARY_EMB_GPU_DTYPE(dtype)                                \
  REGISTER_FUSED_APPLY_ROTARY_EMB_GPU(dtype, int64_t);                                  \
  REGISTER_FUSED_APPLY_ROTARY_EMB_GPU(dtype, int32_t);                                  \
  REGISTER_USER_KERNEL("fused_apply_rotary_emb")                                        \
      .SetCreateFn<FusedApplyRotaryEmbKernel<dtype, int64_t>>()                         \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobInputSize("position_ids") == 0));

REGISTER_FUSED_APPLY_ROTARY_EMB_GPU_DTYPE(float);
REGISTER_FUSED_APPLY_ROTARY_EMB_GPU_DTYPE(half);
#if CUDA_VERSION >= 11000
REGISTER_FUSED_APPLY_ROTARY_EMB_GPU_DTYPE(hip_bfloat16);
#endif  // CUDA_VERSION >= 11000

}  // namespace

}  // namespace oneflow
