#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

void ParseDims(const ShapeView& shape, const std::string& layout,
               const Optional<int64_t>& num_heads, const Optional<int64_t>& head_size,
               int64_t tensor_index, int64_t* b, int64_t* m, int64_t* h, int64_t* k,
               int64_t* b_stride, int64_t* m_stride, int64_t* h_stride, int64_t* offset) {
  if (shape.NumAxes() == 3) {
    if (layout == "BM(HK)" || layout == "BM(H2K)" || layout == "BM(H3K)" || layout == "MB(HK)"
        || layout == "MB(H2K)" || layout == "MB(H3K)") {
      bool batch_first = false;
      int64_t packed_n = 0;
      const std::string layout_bm = layout.substr(0, 2);
      const std::string layout_hk = layout.substr(2);
      if (layout_bm == "BM") {
        *b = shape.At(0);
        *m = shape.At(1);
        batch_first = true;
      } else if (layout_bm == "MB") {
        *b = shape.At(1);
        *m = shape.At(0);
        batch_first = false;
      } else {
        UNIMPLEMENTED();
      }
      if (layout_hk == "(HK)") {
        packed_n = 1;
      } else if (layout_hk == "(H2K)") {
        packed_n = 2;
      } else if (layout_hk == "(H3K)") {
        packed_n = 3;
      } else {
        UNIMPLEMENTED();
      }
      const int64_t hidden_size = shape.At(2);
      if (num_heads) {
        const int64_t expected_h = CHECK_JUST(num_heads);
        const int64_t packed_h = packed_n * expected_h;
        CHECK_EQ(hidden_size % packed_h, 0);
        *h = expected_h;
        *k = hidden_size / packed_h;
      } else if (head_size) {
        const int64_t expected_k = CHECK_JUST(head_size);
        const int64_t packed_k = packed_n * expected_k;
        CHECK_EQ(hidden_size % packed_k, 0);
        *h = hidden_size / packed_k;
        *k = expected_k;
      } else {
        UNIMPLEMENTED();
      }
      *h_stride = *k * packed_n;
      if (batch_first) {
        *m_stride = *h_stride * *h;
        *b_stride = *m_stride * *m;
      } else {
        *b_stride = *h_stride * *h;
        *m_stride = *b_stride * *b;
      }
      if (packed_n == 1) {
        *offset = 0;
      } else if (packed_n == 2) {
        CHECK_GE(tensor_index, 1);
        *offset = (tensor_index - 1) * *k;
      } else if (packed_n == 3) {
        *offset = tensor_index * *k;
      } else {
        UNIMPLEMENTED();
      }
    } else {
      UNIMPLEMENTED();
    }
  } else if (shape.NumAxes() == 4) {
    if (layout == "BMHK") {
      *b = shape.At(0);
      *m = shape.At(1);
      *h = shape.At(2);
      *k = shape.At(3);
      *h_stride = *k;
      *m_stride = *h_stride * *h;
      *b_stride = *m_stride * *m;
    } else if (layout == "BHMK") {
      *b = shape.At(0);
      *m = shape.At(2);
      *h = shape.At(1);
      *k = shape.At(3);
      *m_stride = *k;
      *h_stride = *m_stride * *m;
      *b_stride = *h_stride * *h;
    } else if (layout == "MBHK") {
      *b = shape.At(1);
      *m = shape.At(0);
      *h = shape.At(2);
      *k = shape.At(3);
      *h_stride = *k;
      *b_stride = *h_stride * *h;
      *m_stride = *b_stride * *b;
    } else {
      UNIMPLEMENTED();
    }
    if (num_heads) {
      const int64_t expected_h = CHECK_JUST(num_heads);
      CHECK_EQ(*h, expected_h);
    }
    if (head_size) {
      const int64_t expected_k = CHECK_JUST(head_size);
      CHECK_EQ(*k, expected_k);
    }
    *offset = 0;
  } else {
    UNIMPLEMENTED();
  };
}

template<typename T, typename IndexType, int num_dims>
struct FusedApplyRotaryEmbParam {
  const T* x;
  const T* cos;
  const T* sin;
  const int* position_ids;
  T* out;
  T theta;
  int pass_ndims;
  int rotary_emb_dim; //TODO: dispatch
  IndexType num_elements;
  IndexType k;
  IndexType sinuous_m;
  IndexType offset;
  IndexType x_stride[num_dims];
  IndexType position_stride[num_dims];
  IndexType position_mask[num_dims];
  IndexType sinuous_stride[num_dims];
  IndexType sinuous_mask[num_dims];

//TODO: position_id type? for now it is int32
  FusedApplyRotaryEmbParam(const T* x, const T* cos, const T* sin, const int* position_ids, T* out, const T theta, const int pass_ndims,
                                    const int rotary_emb_dim, const IndexType num_elements, 
                                    const IndexType k, const IndexType sinuous_m,const IndexType offset)
      : x(x), cos(cos), sin(sin), position_ids(position_ids), out(out), theta(theta), pass_ndims(pass_ndims), 
      rotary_emb_dim(rotary_emb_dim), num_elements(num_elements), k(k), sinuous_m(sinuous_m),offset(offset) {}
};

template<typename T, typename IndexType, size_t pack_size, size_t num_dims>
__global__ void FusedApplyRotaryEmbFetchKernel(FusedApplyRotaryEmbParam<T, IndexType, num_dims> param) {
  const T* x = param.x;
  const T* cos = param.cos;
  const T* sin = param.sin;
  const int* position_ids = param.position_ids;
  T* out = param.out;
  const IndexType packed_k = param.k / pack_size;
  const IndexType packed_pass_ndims = param.pass_ndims / pack_size;
  const IndexType packed_rotary_ndims = packed_k - packed_pass_ndims;
  const IndexType rotary_num_elements = param.num_elements / packed_k  * packed_rotary_ndims;
  for (IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x; packed_offset < rotary_num_elements;
       packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset = param.offset + (packed_offset % packed_rotary_ndims + packed_offset / packed_rotary_ndims * packed_k) * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;
    IndexType sinuous_offset = 0;
    IndexType position_id_offset = 0;
    IndexType k_index = 0;

    IndexType temp_offset = offset;
#pragma unloop
    for (int i = 0; i < num_dims; i++) {
      IndexType index = temp_offset / param.x_stride[i];
      if (i == num_dims-1) k_index = index;
      temp_offset = temp_offset - index * param.x_stride[i];
      position_id_offset = position_id_offset + (index * param.position_mask[i] * param.position_stride[i]);
    }

    position_id_offset = position_id_offset + param.sinuous_m * (k_index * param.rotary_emb_dim / (param.k - param.pass_ndims));

    const int position = position_ids[position_id_offset];

    sinuous_offset = position * param.k + k_index;
    
    const LoadPack* cos_load = reinterpret_cast<const LoadPack*>(cos + sinuous_offset);
    const LoadPack* sin_load = reinterpret_cast<const LoadPack*>(sin + sinuous_offset);
    const LoadPack cos_vec = *cos_load, sin_vec = *sin_load;
    LoadPack out_vec;

#pragma unloop
    for (int i = 0; i < pack_size / 2; i++) {
      out_vec.elem[i * 2] =
          x_vec.elem[i * 2] * cos_vec.elem[i * 2] - x_vec.elem[i * 2 + 1] * sin_vec.elem[i * 2];
      out_vec.elem[i * 2 + 1] = x_vec.elem[i * 2 + 1] * cos_vec.elem[i * 2 + 1]
                                + x_vec.elem[i * 2] * sin_vec.elem[i * 2 + 1];
    }

    *(reinterpret_cast<LoadPack*>(out + offset)) = out_vec;
  }


  for(IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x; packed_offset < param.num_elements && packed_offset >= rotary_num_elements; 
      packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset = param.offset + 
      (packed_rotary_ndims + (packed_offset - rotary_num_elements) % packed_pass_ndims 
        + (packed_offset - rotary_num_elements) / packed_pass_ndims * packed_k) * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;

    *(reinterpret_cast<LoadPack*>(out + offset)) = x_vec;
  }
}

template<typename T, typename IndexType, size_t pack_size, size_t num_dims>
__global__ void FusedApplyRotaryEmbComputeKernel(FusedApplyRotaryEmbParam<T, IndexType, num_dims> param) {
  const T* x = param.x;
  const T* cos = param.cos;
  const T* sin = param.sin;
  const int* position_ids = param.position_ids;
  T* out = param.out;
  const T theta = param.theta;
  const IndexType packed_k = param.k / pack_size;
  const IndexType packed_pass_ndims = param.pass_ndims / pack_size;
  const IndexType packed_rotary_ndims = packed_k - packed_pass_ndims;
  const IndexType rotary_num_elements = param.num_elements / packed_k  * packed_rotary_ndims;
  for (IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x; packed_offset < rotary_num_elements;
       packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset = param.offset + (packed_offset % packed_rotary_ndims + packed_offset / packed_rotary_ndims * packed_k) * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;
    IndexType position_id_offset = 0;
    IndexType k_index = 0;

    IndexType temp_offset = offset;
#pragma unloop
    for (int i = 0; i < num_dims; i++) {
      IndexType index = temp_offset / param.x_stride[i];
      if (i == num_dims-1) k_index = index;
      temp_offset = temp_offset - index * param.x_stride[i];
      position_id_offset = position_id_offset + (index * param.position_mask[i] * param.position_stride[i]);
    }

    position_id_offset = position_id_offset + param.sinuous_m * (k_index * param.rotary_emb_dim / (param.k - param.pass_ndims));

    const int position = position_ids[position_id_offset];
    
    LoadPack cos_vec, sin_vec, out_vec;

#pragma unloop
    for (int i = 0; i < pack_size / 2; i++) {
      float val = position * expf(2.0f * static_cast<float>(k_index / 2 + i) / param.k * logf(theta));
      T cos_val = cosf(val);
      T sin_val = sinf(val);
      cos_vec.elem[i*2] = cos_val;
      cos_vec.elem[i*2 + 1] = cos_val;
      sin_vec.elem[i*2] = sin_val;
      sin_vec.elem[i*2 + 1] = sin_val;
    }

#pragma unloop
    for (int i = 0; i < pack_size / 2; i++) {
      out_vec.elem[i * 2] =
          x_vec.elem[i * 2] * cos_vec.elem[i * 2] - x_vec.elem[i * 2 + 1] * sin_vec.elem[i * 2];
      out_vec.elem[i * 2 + 1] = x_vec.elem[i * 2 + 1] * cos_vec.elem[i * 2 + 1]
                                + x_vec.elem[i * 2] * sin_vec.elem[i * 2 + 1];
    }

    *(reinterpret_cast<LoadPack*>(out + offset)) = out_vec;
  }


  for(IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x; packed_offset < param.num_elements && packed_offset >= rotary_num_elements; 
      packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, pack_size>;
    IndexType offset = param.offset + 
      (packed_rotary_ndims + (packed_offset - rotary_num_elements) % packed_pass_ndims 
        + (packed_offset - rotary_num_elements) / packed_pass_ndims * packed_k) * pack_size;
    const LoadPack* x_load = reinterpret_cast<const LoadPack*>(x + offset);
    const LoadPack x_vec = *x_load;

    *(reinterpret_cast<LoadPack*>(out + offset)) = x_vec;
  }
}

template<typename T, typename IndexType, size_t pack_size, size_t num_dims>
void LaunchKernel(const T* x, const T* cos, const T* sin, const int* position_ids, T* out, const int64_t* position_shape, const std::string& layout, 
                    const T theta, const int pass_ndims, const int rotary_emb_dim, const int64_t b,
                    const int64_t m, const int64_t h, const int64_t k, const int64_t b_stride, 
                    const int64_t m_stride, const int64_t h_stride, const int64_t offset, 
                    IndexType num_elements) {
  DimVector kernel_x_shape(num_dims), kernel_sinuous_shape(num_dims);
  size_t x_stride[num_dims];
  size_t sinuous_stride[num_dims];

  x_stride[num_dims - 1] = 1;
  sinuous_stride[num_dims - 1] = 1;

  for (int i = num_dims - 2; i >= 0; i--) {
    x_stride[i] = x_stride[i + 1] * kernel_x_shape.at(i + 1);
    sinuous_stride[i] = sinuous_stride[i + 1] * kernel_sinuous_shape.at(i + 1);
  }

  struct FusedApplyRotaryEmbParam<T, IndexType, num_dims> param(
      reinterpret_cast<const T*>(x), reinterpret_cast<const T*>(cos),
      reinterpret_cast<const T*>(sin), reinterpret_cast<const int*>(position_ids), reinterpret_cast<T*>(out), theta, pass_ndims, rotary_emb_dim,
      num_elements, k, static_cast<IndexType>(position_shape[2]), offset);

  std::pair<char, std::int64_t> strides[num_dims];
  strides[0] = {'b', b_stride};
  strides[1] = {'h', h_stride};
  strides[2] = {'m', m_stride};
  strides[3] = {'k', 1};

  auto GetDimX = [&](const char c) {
    if (c == 'b') {
        return b;
    } else if (c == 'h') {
        return h;
    } else if (c == 'm') {
        return m;
    } else if (c == 'k') {
        return k;
    }

    return 0L;
  };

  std::sort(strides, strides + num_dims, [&](auto pair1, auto pair2) {
    if (pair1.second > pair2.second) {
        return true;
    } else if (pair1.second == pair2.second) {
        if (GetDimX(pair1.first) != 1) {
            return true;
        }
        return false;
    } else {
        return false;
    }
    return pair1.second > pair2.second;
  });

// K has to be the last dimension, only k&m matters, therefore strides other than k&m does not really needs to be computed
#pragma unloop
  for (int i = 0; i < num_dims; i++) {
    param.x_stride[i] = strides[i].second;
    param.sinuous_mask[i] = 0;
    param.position_mask[i] = 0;
    if (strides[i].first == 'm') {
      param.sinuous_mask[i] = 1;
      param.sinuous_stride[i] = k;
      param.position_stride[i] = 1;
      param.position_mask[i] = 1;
    } else if (strides[i].first == 'k') {
      param.sinuous_mask[i] = 1;
      param.sinuous_stride[i] = 1;
    } else if (strides[i].first == 'b') {
      param.position_stride[i] = rotary_emb_dim * position_shape[2];
      param.position_mask[i] = 1;
    }
  }

  constexpr size_t blk_size = 128;

  if (cos) {
    FusedApplyRotaryEmbFetchKernel<T, IndexType, pack_size, num_dims>
        <<<(param.num_elements + blk_size - 1) / blk_size, blk_size>>>(param);
  } else {
    FusedApplyRotaryEmbComputeKernel<T, IndexType, pack_size, num_dims>
        <<<(param.num_elements + blk_size - 1) / blk_size, blk_size>>>(param);
  }
}

template<typename T, typename IndexType, size_t num_dims>
void DispatchPackSize(const T* x, const T* cos, const T* sin, const int* position_ids, T* out, const int64_t* position_shape, const std::string& layout, 
                      const T theta, const int pass_ndims, const int rotary_emb_dim, const IndexType b, 
                      const IndexType m, const IndexType h, const IndexType k, const IndexType b_stride,
                      const IndexType m_stride, const IndexType h_stride, const IndexType offset, 
                      IndexType num_elements) {
  const auto CheckPackSize = [&](const size_t pack_size) {
    bool r = (((reinterpret_cast<uintptr_t>(x) % (sizeof(T) * pack_size)) == 0)
              && ((((k - pass_ndims) / 2) % pack_size) == 0) && ((pass_ndims % pack_size) == 0) && ((16 / sizeof(T)) >= pack_size));
    return r;
  };

  if (CheckPackSize(8)) {
    num_elements /= 8;
    LaunchKernel<T, IndexType, 8, num_dims>(x, cos, sin, position_ids, out, position_shape, layout, theta, pass_ndims, rotary_emb_dim, b, m, h, k, 
                                          b_stride, m_stride, h_stride, offset, num_elements);
  } else if (CheckPackSize(4)) {
    num_elements /= 4;
    LaunchKernel<T, IndexType, 4, num_dims>(x, cos, sin, position_ids, out, position_shape, layout, theta, pass_ndims, rotary_emb_dim, b, m, h, k, 
                                          b_stride, m_stride, h_stride, offset, num_elements);
  } else {
    num_elements /= 2;
    LaunchKernel<T, IndexType, 2, num_dims>(x, cos, sin, position_ids, out, position_shape, layout, theta, pass_ndims, rotary_emb_dim, b, m, h, k, 
                                          b_stride, m_stride, h_stride, offset, num_elements);
  }
}

template<typename T, size_t num_dims>
void DispatchIndex(const T* x, const T* cos, const T* sin, const int* position_ids, T* out, const int64_t* position_shape, const std::string& layout, 
  const T theta, const int pass_ndims, const int rotary_emb_dim, const int64_t b, const int64_t m, const int64_t h, const int64_t k, const int64_t b_stride, 
  const int64_t m_stride, const int64_t h_stride, const int64_t offset) {
  int64_t num_elements = b * m * h * k;
  if (num_elements < (1 << 30)) {
    DispatchPackSize<T, int32_t, num_dims>(x, cos, sin, position_ids, out, position_shape, layout, theta, pass_ndims, rotary_emb_dim,static_cast<int32_t>(b), 
                                          static_cast<int32_t>(m), static_cast<int32_t>(h), static_cast<int32_t>(k), 
                                          static_cast<int32_t>(b_stride), static_cast<int32_t>(m_stride), 
                                          static_cast<int32_t>(h_stride), static_cast<int32_t>(offset), 
                                          static_cast<int32_t>(num_elements));
  } else {
    DispatchPackSize<T, int64_t, num_dims>(x, cos, sin, position_ids, out, position_shape, layout, theta, pass_ndims, rotary_emb_dim, b, m, h, k, 
                                          b_stride, m_stride, h_stride, offset, num_elements);
  }
}

template<typename T>
class FusedApplyRotaryEmbKernel final : public user_op::OpKernel {
 public:
  FusedApplyRotaryEmbKernel() = default;
  ~FusedApplyRotaryEmbKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    auto* hip_stream = ctx->stream()->As<ep::CudaStream>();

    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* cos =nullptr;
    user_op::Tensor* sin = nullptr;
    user_op::Tensor* position_ids = nullptr;
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const std::string& layout = ctx->Attr<std::string>("layout");
    const int pass_ndims = ctx->Attr<int>("pass_ndims");
    const float theta = ctx->Attr<float>("theta");
    int rotary_emb_dim = 1;

    if (ctx->has_input("cos", 0)) {
      cos = ctx->Tensor4ArgNameAndIndex("cos", 0);
    }

    if (ctx->has_input("sin", 0)) {
      sin = ctx->Tensor4ArgNameAndIndex("sin", 0);
    }

    if (ctx->has_input("position_ids", 0)) {
      position_ids = ctx->Tensor4ArgNameAndIndex("position_ids", 0);
      rotary_emb_dim = position_ids->shape_view().At(1);
    }

    constexpr size_t N = 4;
    int64_t b = 0;
    int64_t m = 0;
    int64_t h = 0;
    int64_t k = 0;
    int64_t b_stride = 0;
    int64_t m_stride = 0;
    int64_t h_stride = 0;
    int64_t offset   = 0;
    
    ParseDims(x->shape_view(), layout, Optional<int64_t>(), cos ? Optional<int64_t>(cos->shape_view().At(1)) : Optional<int64_t>(), 1,
      &b, &m, &h, &k, &b_stride, &m_stride, &h_stride, &offset);

    // TODO: hard code num_dims & seems redundant template problem...
    DispatchIndex<T, N>(
        reinterpret_cast<const T*>(x->dptr()), cos ? reinterpret_cast<const T*>(cos->dptr()) : nullptr,
        sin ? reinterpret_cast<const T*>(sin->dptr()) : nullptr, position_ids ? reinterpret_cast<const int*>(position_ids->dptr()) :
          nullptr, reinterpret_cast<T*>(out->mut_dptr()), position_ids ? position_ids->shape_view().data() : nullptr, 
          layout, static_cast<T>(theta), pass_ndims, rotary_emb_dim, b, m, h, k, b_stride, m_stride, h_stride, offset);
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_APPLY_ROTARY_EMB_GPU(dtype)              \
  REGISTER_USER_KERNEL("fused_apply_rotary_emb")                       \
      .SetCreateFn<FusedApplyRotaryEmbKernel<dtype>>()                \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_APPLY_ROTARY_EMB_GPU(float);
REGISTER_FUSED_APPLY_ROTARY_EMB_GPU(half);
#if CUDA_VERSION >= 11000
REGISTER_FUSED_APPLY_ROTARY_EMB_GPU(hip_bfloat16);
#endif  // CUDA_VERSION >= 11000

}  // namespace

}  // namespace oneflow
