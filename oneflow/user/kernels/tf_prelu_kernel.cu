#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ndarray/ndarray_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void BroadcastPReluForwardGpu(const int32_t elem_cnt, const int32_t alpha_size,
                                         const int32_t inner_size, const T* x, const T* alpha,
                                         T* y) {
  T zero_val = static_cast<T>(0.0);
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const T x_i = x[i];
    const T alpha_i = alpha[(i / inner_size) % alpha_size];
    y[i] = x_i > zero_val ? x_i : x_i * alpha_i;
  }
}

template<typename T>
__global__ void BroadcastPReluBackwardGpu(const int32_t elem_cnt, const int32_t alpha_size,
                                          const int32_t inner_size, const T* x, const T* alpha,
                                          const T* dy, T* dx, T* alpha_diff) {
  T zero_val = static_cast<T>(0.0);
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const T x_i = x[i];
    const T dy_i = dy[i];
    const T alpha_i = alpha[(i / inner_size) % alpha_size];
    T dx_i = zero_val;
    T alpha_diff_i = zero_val;
    if (x_i > zero_val) {
      dx_i = dy_i;
      alpha_diff_i = zero_val;
    } else {
      dx_i = dy_i * alpha_i;
      alpha_diff_i = dy_i * x_i;
    }
    dx[i] = dx_i;
    alpha_diff[i] = alpha_diff_i;
  }
}

template<typename T>
__global__ void ElemwisePReluForwardGpu(const int32_t elem_cnt, const T* x, const T* alpha, T* y) {
  T zero_val = static_cast<T>(0.0);
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const T x_i = x[i];
    const T alpha_i = alpha[i];
    y[i] = x_i > zero_val ? x_i : x_i * alpha_i;
  }
}

template<typename T>
__global__ void ElemwisePReluBackwardGpu(const int32_t elem_cnt, const T* x, const T* alpha,
                                         const T* dy, T* dx, T* alpha_diff) {
  T zero_val = static_cast<T>(0.0);
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const T x_i = x[i];
    const T dy_i = dy[i];
    const T alpha_i = alpha[i];
    T dx_i = zero_val;
    T alpha_diff_i = zero_val;
    if (x_i > zero_val) {
      dx_i = dy_i;
      alpha_diff_i = zero_val;
    } else {
      dx_i = dy_i * alpha_i;
      alpha_diff_i = dy_i * x_i;
    }
    dx[i] = dx_i;
    alpha_diff[i] = alpha_diff_i;
  }
}

bool IsAlphaShapeContiguous(const ShapeView& alpha_shape, const ShapeView& x_shape) {
  if (alpha_shape.elem_cnt() == 1) { return true; }
  int64_t begin_idx = -1;
  for (int64_t i = 0; i < alpha_shape.NumAxes(); ++i) {
    if (alpha_shape.At(i) != 1) {
      begin_idx = i;
      break;
    }
  }
  CHECK_NE(begin_idx, -1);
  int64_t end_idx = -1;
  for (int64_t i = alpha_shape.NumAxes(); i > 0; --i) {
    if (alpha_shape.At(i - 1) != 1) {
      end_idx = i;
      break;
    }
  }
  CHECK_NE(end_idx, -1);
  if (alpha_shape.elem_cnt() == x_shape.Count(begin_idx + 1, end_idx + 1)) {
    return true;
  } else {
    return false;
  }
}

int32_t GetOuterSize(const ShapeView& alpha_shape, const ShapeView& x_shape) {
  int32_t outer_size = x_shape.At(0);
  for (int32_t i = 0; i < alpha_shape.NumAxes(); ++i) {
    if (alpha_shape.At(i) == 1) {
      outer_size *= x_shape.At(i + 1);
    } else {
      break;
    }
  }
  return outer_size;
}

}  // namespace

template<typename T>
class TfGpuPReluKernel final : public user_op::OpKernel {
 public:
  TfGpuPReluKernel() = default;
  ~TfGpuPReluKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* alpha = ctx->Tensor4ArgNameAndIndex("alpha", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const int32_t elem_cnt = x->shape_view().elem_cnt();
    if (IsAlphaShapeContiguous(alpha->shape_view(), x->shape_view())) {
      const int32_t outer_size = GetOuterSize(alpha->shape_view(), x->shape_view());
      const int32_t alpha_size = alpha->shape_view().elem_cnt();
      const int32_t inner_size = elem_cnt / outer_size / alpha_size;
      BroadcastPReluForwardGpu<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                    ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          elem_cnt, alpha_size, inner_size, x->dptr<T>(), alpha->dptr<T>(), y->mut_dptr<T>());
    } else {
      user_op::Tensor* broadcasted_alpha = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
      const Shape& left_extended_shape =
          CreateLeftExtendedShape(ShapeView(alpha->shape_view()), x->shape_view().NumAxes());
      NdarrayUtil<DeviceType::kCUDA, T>::BroadcastTo(
          ctx->stream(), XpuVarNdarray<T>(x->shape_view(), broadcasted_alpha->mut_dptr<T>()),
          XpuVarNdarray<const T>(left_extended_shape, alpha->dptr<T>()));
      ElemwisePReluForwardGpu<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                   ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          elem_cnt, x->dptr<T>(), broadcasted_alpha->dptr<T>(), y->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_TF_CUDA_PRELU_KERNEL(dtype)                                           \
  REGISTER_USER_KERNEL("tf_prelu")                                                     \
      .SetCreateFn<TfGpuPReluKernel<dtype>>()                                          \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                 \
                       && (user_op::HobDataType("y", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                              \
        const Shape& in_shape = ctx->InputShape("x", 0);                               \
        const Shape& alpha_shape = ctx->InputShape("alpha", 0);                        \
        const int64_t tmp_buffer_size =                                                \
            IsAlphaShapeContiguous(alpha_shape, in_shape)                              \
                ? 0                                                                    \
                : GetCudaAlignedSize(in_shape.elem_cnt() * sizeof(dtype));             \
        return tmp_buffer_size;                                                        \
      });

REGISTER_TF_CUDA_PRELU_KERNEL(half)
REGISTER_TF_CUDA_PRELU_KERNEL(float)
REGISTER_TF_CUDA_PRELU_KERNEL(double)

template<typename T>
class TfGpuPReluGradKernel final : public user_op::OpKernel {
 public:
  TfGpuPReluGradKernel() = default;
  ~TfGpuPReluGradKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* alpha = ctx->Tensor4ArgNameAndIndex("alpha", 0);
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    user_op::Tensor* alpha_diff = ctx->Tensor4ArgNameAndIndex("alpha_diff", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int32_t elem_cnt = x->shape_view().elem_cnt();
    T* broadcasted_alpha_diff = tmp_buffer->mut_dptr<T>();
    T* reduce_sum_tmp_buf = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>()
                                                 + GetCudaAlignedSize(elem_cnt * sizeof(T)));
    const Shape& left_extended_shape =
        CreateLeftExtendedShape(ShapeView(alpha->shape_view()), x->shape_view().NumAxes());
    if (IsAlphaShapeContiguous(alpha->shape_view(), x->shape_view())) {
      const int32_t outer_size = GetOuterSize(alpha->shape_view(), x->shape_view());
      const int32_t alpha_size = alpha->shape_view().elem_cnt();
      const int32_t inner_size = elem_cnt / outer_size / alpha_size;
      BroadcastPReluBackwardGpu<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                     ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          elem_cnt, alpha_size, inner_size, x->dptr<T>(), alpha->dptr<T>(), dy->dptr<T>(),
          dx->mut_dptr<T>(), broadcasted_alpha_diff);
    } else {
      T* broadcasted_alpha = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>()
                                                  + 2 * GetCudaAlignedSize(elem_cnt * sizeof(T)));

      NdarrayUtil<DeviceType::kCUDA, T>::BroadcastTo(
          ctx->stream(), XpuVarNdarray<T>(x->shape_view(), broadcasted_alpha),
          XpuVarNdarray<const T>(left_extended_shape, alpha->dptr<T>()));

      ElemwisePReluBackwardGpu<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                    ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          elem_cnt, x->dptr<T>(), broadcasted_alpha, dy->dptr<T>(), dx->mut_dptr<T>(),
          broadcasted_alpha_diff);
    }
    NdarrayUtil<DeviceType::kCUDA, T>::ReduceSum(
        ctx->stream(), XpuVarNdarray<T>(left_extended_shape, alpha_diff->mut_dptr<T>()),
        XpuVarNdarray<const T>(x->shape_view(), broadcasted_alpha_diff),
        XpuVarNdarray<T>(x->shape_view(), reduce_sum_tmp_buf));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_TF_CUDA_PRELU_GRAD_KERNEL(dtype)                                       \
  REGISTER_USER_KERNEL("tf_prelu_grad")                                                 \
      .SetCreateFn<TfGpuPReluGradKernel<dtype>>()                                       \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value)) \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                               \
        const Shape& in_shape = ctx->InputShape("x", 0);                                \
        const Shape& alpha_shape = ctx->InputShape("alpha", 0);                         \
        const int64_t tmp_buffer_size =                                                 \
            IsAlphaShapeContiguous(alpha_shape, in_shape)                               \
                ? 2 * GetCudaAlignedSize(in_shape.elem_cnt() * sizeof(dtype))           \
                : 3 * GetCudaAlignedSize(in_shape.elem_cnt() * sizeof(dtype));          \
        return tmp_buffer_size;                                                         \
      });

REGISTER_TF_CUDA_PRELU_GRAD_KERNEL(half)
REGISTER_TF_CUDA_PRELU_GRAD_KERNEL(float)
REGISTER_TF_CUDA_PRELU_GRAD_KERNEL(double)

}  // namespace oneflow
