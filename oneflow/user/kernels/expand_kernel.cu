#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/user/kernels/expand_kernel_utils.h"

namespace oneflow {

namespace {

const int32_t NDIMS = 16;
struct STRIDES {
  int32_t val[NDIMS];
};

template<typename T>
__global__ void ExpandCudaKernel(const T* in_ptr, const STRIDES in_stride,
                                 const STRIDES expand_stride, const int32_t dims,
                                 const int32_t elements, T* out_ptr) {
  int32_t gid = (blockDim.x * blockIdx.x) + threadIdx.x;
  int32_t step = gridDim.x * blockDim.x;
  while (gid < elements) {
    int32_t offset = OffsetToNdIndexToOffset(gid, in_stride.val, expand_stride.val, dims);
    out_ptr[gid] = in_ptr[offset];
    gid += step;
  }
}

template<typename T>
__global__ void ExpandGradCudaKernel(const T* out_diff_ptr, const STRIDES out_stride,
                                     const STRIDES expand_stride, const int32_t dims,
                                     const int32_t elements, T* in_diff_ptr) {
  int32_t gid = (blockDim.x * blockIdx.x) + threadIdx.x;
  int32_t step = gridDim.x * blockDim.x;
  while (gid < elements) {
    int32_t offset = OffsetToNdIndexToOffset(gid, out_stride.val, expand_stride.val, dims);
    cuda::atomic::Add(&in_diff_ptr[offset], out_diff_ptr[gid]);
    gid += step;
  }
}

template<typename T>
__global__ void InitPtr(const int32_t elements, T* ptr) {
  int32_t gid = (blockDim.x * blockIdx.x) + threadIdx.x;
  int32_t step = gridDim.x * blockDim.x;
  while (gid < elements) {
    ptr[gid] = static_cast<T>(0);
    gid += step;
  }
}

template<typename T>
struct GpuExpandFunctor final {
  void operator()(ep::Stream* stream, const T* in_ptr, const STRIDES in_stride,
                  const STRIDES expand_stride, const int32_t dims, const int32_t elements,
                  T* out_ptr) {
    RUN_CUDA_KERNEL((ExpandCudaKernel<T>), stream, elements, in_ptr, in_stride, expand_stride, dims,
                    elements, out_ptr);
  }
};

template<>
void GpuExpandFunctor<float16>::operator()(ep::Stream* stream, const float16* in_ptr,
                                           const STRIDES in_stride, const STRIDES expand_stride,
                                           const int32_t dims, const int32_t elements,
                                           float16* out_ptr) {
  RUN_CUDA_KERNEL((ExpandCudaKernel<half>), stream, elements, reinterpret_cast<const half*>(in_ptr),
                  in_stride, expand_stride, dims, elements, reinterpret_cast<half*>(out_ptr));
}

template<typename T>
struct GpuExpandGradFunctor final {
  void operator()(ep::Stream* stream, const T* in_ptr, const STRIDES in_stride,
                  const STRIDES expand_stride, const int32_t dims, const int32_t elements,
                  const int32_t out_elements, T* out_ptr) {
    RUN_CUDA_KERNEL((InitPtr<T>), stream, out_elements, out_elements, out_ptr);
    RUN_CUDA_KERNEL((ExpandGradCudaKernel<T>), stream, elements, in_ptr, in_stride, expand_stride,
                    dims, elements, out_ptr);
  }
};

template<>
void GpuExpandGradFunctor<float16>::operator()(ep::Stream* stream, const float16* in_ptr,
                                               const STRIDES in_stride, const STRIDES expand_stride,
                                               const int32_t dims, const int32_t elements,
                                               const int32_t out_elements, float16* out_ptr) {
  RUN_CUDA_KERNEL((InitPtr<half>), stream, out_elements, out_elements,
                  reinterpret_cast<half*>(out_ptr));
  RUN_CUDA_KERNEL((ExpandGradCudaKernel<half>), stream, elements,
                  reinterpret_cast<const half*>(in_ptr), in_stride, expand_stride, dims, elements,
                  reinterpret_cast<half*>(out_ptr));
}

}  // namespace

template<typename T>
class GpuExpandKernel final : public user_op::OpKernel {
 public:
  GpuExpandKernel() = default;
  ~GpuExpandKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const std::vector<int32_t>& logical_expand_shape =
        ctx->Attr<std::vector<int32_t>>("logical_expand_shape");
    if (std::any_of(logical_expand_shape.begin(), logical_expand_shape.end(),
                    [](int32_t dim_size) { return dim_size == 0; })) {
      return;
    }
    std::vector<int32_t> in_shape;
    in_shape.resize(in->shape().NumAxes());
    for (int i = 0; i < in->shape().NumAxes(); ++i) { in_shape[i] = in->shape().At(i); }

    std::vector<int32_t> out_shape;
    std::vector<int32_t> stride;
    CHECK_JUST(getOutShapeAndStrideForFp(in_shape, logical_expand_shape, out_shape, stride));

    const T* in_ptr = in->dptr<T>();
    T* out_ptr = out->mut_dptr<T>();
    const int32_t out_dims = out->shape().NumAxes();
    const int32_t out_size = out->shape().elem_cnt();

    STRIDES expand_stride;
    for (int i = 0; i < out_dims; ++i) { expand_stride.val[i] = stride[i]; }
    STRIDES out_stride;
    InitStride(out_stride.val, out_shape.data(), out_dims);
    GpuExpandFunctor<T>()(ctx->stream(), in_ptr, out_stride, expand_stride, out_dims, out_size,
                          out_ptr);
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_EXPAND_KERNEL(dtype)                                                   \
  REGISTER_USER_KERNEL("expand").SetCreateFn<GpuExpandKernel<dtype>>().SetIsMatchedHob( \
      (user_op::HobDeviceType() == DeviceType::kCUDA)                                   \
      && (user_op::HobDataType("in", 0) == GetDataType<dtype>::value))

REGISTER_EXPAND_KERNEL(float);
REGISTER_EXPAND_KERNEL(double);
REGISTER_EXPAND_KERNEL(float16);
REGISTER_EXPAND_KERNEL(bool);
REGISTER_EXPAND_KERNEL(uint8_t);
REGISTER_EXPAND_KERNEL(int8_t);
REGISTER_EXPAND_KERNEL(int32_t);
REGISTER_EXPAND_KERNEL(int64_t);

template<typename T>
class GpuExpandGradKernel final : public user_op::OpKernel {
 public:
  GpuExpandGradKernel() = default;
  ~GpuExpandGradKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const std::vector<int32_t>& logical_out_shape =
        ctx->Attr<std::vector<int32_t>>("logical_out_shape");
    const std::vector<int32_t>& logical_expand_shape =
        ctx->Attr<std::vector<int32_t>>("logical_expand_shape");

    std::vector<int32_t> in_shape;
    in_shape.resize(in->shape().NumAxes());
    for (int i = 0; i < in->shape().NumAxes(); ++i) { in_shape[i] = in->shape().At(i); }
    std::vector<int32_t> out_shape;
    std::vector<int32_t> stride;
    CHECK_JUST(getOutShapeAndStrideForBp(logical_out_shape, logical_expand_shape, in_shape,
                                         out_shape, stride));

    const T* in_ptr = in->dptr<T>();
    T* out_ptr = out->mut_dptr<T>();

    const int32_t in_dims = in->shape().NumAxes();
    const int32_t in_size = in->shape().elem_cnt();
    const int32_t out_size = out->shape().elem_cnt();

    STRIDES expand_stride;
    for (int i = 0; i < in_dims; ++i) { expand_stride.val[i] = stride[i]; }
    STRIDES in_stride;
    InitStride(in_stride.val, in_shape.data(), in_dims);

    GpuExpandGradFunctor<T>()(ctx->stream(), in_ptr, in_stride, expand_stride, in_dims, in_size,
                              out_size, out_ptr);
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_EXPAND_GRAD_KERNEL(dtype)                             \
  REGISTER_USER_KERNEL("expand_grad")                                  \
      .SetCreateFn<GpuExpandGradKernel<dtype>>()                       \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("in", 0) == GetDataType<dtype>::value))

REGISTER_EXPAND_GRAD_KERNEL(float);
REGISTER_EXPAND_GRAD_KERNEL(double);
REGISTER_EXPAND_GRAD_KERNEL(float16);
REGISTER_EXPAND_GRAD_KERNEL(int32_t);
REGISTER_EXPAND_GRAD_KERNEL(int64_t);

}  // namespace oneflow
