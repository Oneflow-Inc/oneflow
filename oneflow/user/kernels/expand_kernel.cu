#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/user/kernels/expand_kernel_utils.h"

namespace oneflow {

namespace {

const int32_t NDIMS = 16;
struct STRIDES {
  int32_t val[NDIMS];
};

template<typename T>
__global__ void ExpandCudaKernel(const T* in_ptr, const STRIDES in_stride,
                                 const STRIDES expand_stride, const int32_t dims,
                                 const int32_t elements, T* out_ptr) {
  int32_t gid = (blockDim.x * blockIdx.x) + threadIdx.x;
  int32_t step = gridDim.x * blockDim.x;
  while (gid < elements) {
    int32_t offset = OffsetToNdIndexToOffset(gid, in_stride.val, expand_stride.val, dims);
    out_ptr[gid] = in_ptr[offset];
    gid += step;
  }
}

template<typename T>
__global__ void ExpandGradCudaKernel(const T* out_diff_ptr, const STRIDES out_stride,
                                     const STRIDES expand_stride, const int32_t dims,
                                     const int32_t elements, T* in_diff_ptr) {
  int32_t gid = (blockDim.x * blockIdx.x) + threadIdx.x;
  int32_t step = gridDim.x * blockDim.x;
  while (gid < elements) {
    int32_t offset = OffsetToNdIndexToOffset(gid, out_stride.val, expand_stride.val, dims);
    cuda::atomic::Add(&in_diff_ptr[offset], out_diff_ptr[gid]);
    gid += step;
  }
}

template<typename T>
__global__ void InitPtr(const int32_t elements, T* ptr) {
  int32_t gid = (blockDim.x * blockIdx.x) + threadIdx.x;
  int32_t step = gridDim.x * blockDim.x;
  while (gid < elements) {
    ptr[gid] = static_cast<T>(0);
    gid += step;
  }
}

template<typename T>
struct GpuExpandFunctor final {
  void operator()(DeviceCtx* ctx, const T* in_ptr, const STRIDES in_stride,
                  const STRIDES expand_stride, const int32_t dims, const int32_t elements,
                  T* out_ptr) {
    RUN_CUDA_KERNEL((ExpandCudaKernel<T>), ctx, elements, in_ptr, in_stride, expand_stride, dims,
                    elements, out_ptr);
  }
};

template<>
void GpuExpandFunctor<float16>::operator()(DeviceCtx* ctx, const float16* in_ptr,
                                           const STRIDES in_stride, const STRIDES expand_stride,
                                           const int32_t dims, const int32_t elements,
                                           float16* out_ptr) {
  RUN_CUDA_KERNEL((ExpandCudaKernel<half>), ctx, elements, reinterpret_cast<const half*>(in_ptr),
                  in_stride, expand_stride, dims, elements, reinterpret_cast<half*>(out_ptr));
}

template<typename T>
struct GpuExpandGradFunctor final {
  void operator()(DeviceCtx* ctx, const T* in_ptr, const STRIDES in_stride,
                  const STRIDES expand_stride, const int32_t dims, const int32_t elements,
                  const int32_t out_elements, T* out_ptr) {
    RUN_CUDA_KERNEL((InitPtr<T>), ctx, out_elements, out_elements, out_ptr);
    RUN_CUDA_KERNEL((ExpandGradCudaKernel<T>), ctx, elements, in_ptr, in_stride, expand_stride,
                    dims, elements, out_ptr);
  }
};

template<>
void GpuExpandGradFunctor<float16>::operator()(DeviceCtx* ctx, const float16* in_ptr,
                                               const STRIDES in_stride, const STRIDES expand_stride,
                                               const int32_t dims, const int32_t elements,
                                               const int32_t out_elements, float16* out_ptr) {
  RUN_CUDA_KERNEL((InitPtr<half>), ctx, out_elements, out_elements,
                  reinterpret_cast<half*>(out_ptr));
  RUN_CUDA_KERNEL((ExpandGradCudaKernel<half>), ctx, elements,
                  reinterpret_cast<const half*>(in_ptr), in_stride, expand_stride, dims, elements,
                  reinterpret_cast<half*>(out_ptr));
}

}  // namespace

template<typename T>
class GpuExpandKernel final : public user_op::OpKernel {
 public:
  GpuExpandKernel() = default;
  ~GpuExpandKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);

    const std::vector<int32_t> stride = ctx->Attr<std::vector<int32_t>>("stride");

    const T* in_ptr = in->dptr<T>();
    T* out_ptr = out->mut_dptr<T>();
    const int32_t out_dims = out->shape().NumAxes();
    const int32_t out_size = out->shape().elem_cnt();

    STRIDES expand_stride;
    for (int i = 0; i < out_dims; ++i) { expand_stride.val[i] = stride[i]; }
    DimVector out_dim_vec;
    out->shape().ToDimVector(&out_dim_vec);
    STRIDES out_stride;
    InitStride(out_stride.val, out_dim_vec.data(), out_dims);
    GpuExpandFunctor<T>()(ctx->device_ctx(), in_ptr, out_stride, expand_stride, out_dims, out_size,
                          out_ptr);
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_EXPAND_KERNEL(dtype)                                                   \
  REGISTER_USER_KERNEL("expand").SetCreateFn<GpuExpandKernel<dtype>>().SetIsMatchedHob( \
      (user_op::HobDeviceTag() == DeviceType::kGPU)                                     \
      & (user_op::HobDataType("in", 0) == GetDataType<dtype>::value))

REGISTER_EXPAND_KERNEL(float);
REGISTER_EXPAND_KERNEL(double);
REGISTER_EXPAND_KERNEL(float16);
REGISTER_EXPAND_KERNEL(int);

template<typename T>
class GpuExpandGradKernel final : public user_op::OpKernel {
 public:
  GpuExpandGradKernel() = default;
  ~GpuExpandGradKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);

    const std::vector<int32_t> stride = ctx->Attr<std::vector<int32_t>>("stride");

    const T* in_ptr = in->dptr<T>();
    T* out_ptr = out->mut_dptr<T>();

    const int32_t in_dims = in->shape().NumAxes();
    const int32_t in_size = in->shape().elem_cnt();
    const int32_t out_size = out->shape().elem_cnt();

    STRIDES expand_stride;
    for (int i = 0; i < in_dims; ++i) { expand_stride.val[i] = stride[i]; }
    DimVector in_dim_vec;
    in->shape().ToDimVector(&in_dim_vec);
    STRIDES in_stride;
    InitStride(in_stride.val, in_dim_vec.data(), in_dims);
    GpuExpandGradFunctor<T>()(ctx->device_ctx(), in_ptr, in_stride, expand_stride, in_dims, in_size,
                              out_size, out_ptr);
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_EXPAND_GRAD_KERNEL(dtype)                           \
  REGISTER_USER_KERNEL("expand_grad")                                \
      .SetCreateFn<GpuExpandGradKernel<dtype>>()                     \
      .SetIsMatchedHob((user_op::HobDeviceTag() == DeviceType::kGPU) \
                       & (user_op::HobDataType("in", 0) == GetDataType<dtype>::value))

REGISTER_EXPAND_GRAD_KERNEL(float);
REGISTER_EXPAND_GRAD_KERNEL(double);
REGISTER_EXPAND_GRAD_KERNEL(float16);
REGISTER_EXPAND_GRAD_KERNEL(int);

}  // namespace oneflow
