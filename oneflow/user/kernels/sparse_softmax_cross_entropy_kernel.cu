#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ndarray/ndarray_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/sparse_cross_entropy_kernel_util.h"
#include "oneflow/user/kernels/sparse_softmax_cross_entropy_kernel_util.h"
#include "oneflow/core/cuda/softmax.cuh"

namespace oneflow {
namespace user_op {

namespace {

template<typename T, typename K>
__global__ void ComputeResultGpu(const int64_t n, const int64_t w, const int64_t depth,
                                 const int64_t lower_bound, const K* labels, T* tmp, T* new_tmp,
                                 T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    assert(labels[i] >= 0);
    assert(labels[i] < depth);
    K label = labels[i] - lower_bound;
    if (label >= 0 && label < w) { y[i] = SafeLog(tmp[i]) - new_tmp[i * w + label]; }
  }
}

template<typename K>
__global__ void ComputeResultGpuHalf(const int64_t n, const int64_t w, const int64_t depth,
                                     const int64_t lower_bound, const K* labels, half* tmp,
                                     half* new_tmp, half* y) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) {
    assert(labels[i] >= 0);
    assert(labels[i] < depth);
    K label = labels[i] - lower_bound;
    if (label >= 0 && label < w) {
      y[i] = __float2half(SafeLog(__half2float(tmp[i])) - __half2float(new_tmp[i * w + label]));
    }
  }
#else
  printf("use half need nvcc arch >= 530");
  assert(false);
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)*/
}

template<typename T>
size_t GetReduceTempStorageSize(int64_t n, int64_t w) {
  return GetCudaAlignedSize(n * w * sizeof(T));
}
template<typename T>
size_t GetProbStorageSize(int64_t n, int64_t w) {
  return GetCudaAlignedSize(n * sizeof(T));
}
template<typename T>
void ComputeProb(DeviceCtx* ctx, const int64_t row, const int64_t col, const T* in, T* prob,
                 T* sub_result, T* sum_result) {
  using ComputeType = typename cuda::softmax::DefaultComputeType<T>::type;
  cuda::softmax::DirectLoad<T, ComputeType> load(in, col);
  cuda::softmax::DirectStore<ComputeType, T> store(prob, col);
  cuda::softmax::DirectStore<ComputeType, T> sub_result_store(sub_result, col);
  cuda::softmax::DirectStore<ComputeType, T> sum_result_store(sum_result, row);
  cuda::softmax::DispatchSoftmax<decltype(load), decltype(store), ComputeType>(
      ctx->hip_stream(), load, store, row, col, sub_result_store, sum_result_store);
}

template<>
void ComputeProb(DeviceCtx* ctx, const int64_t row, const int64_t col, const float16* in,
                 float16* prob, float16* sub_result, float16* sum_result) {
  cuda::softmax::DirectLoad<half, float> load(reinterpret_cast<const half*>(in), col);
  cuda::softmax::DirectStore<float, half> store(reinterpret_cast<half*>(prob), col);
  cuda::softmax::DirectStore<float, half> sub_result_store(reinterpret_cast<half*>(sub_result),
                                                           col);
  cuda::softmax::DirectStore<float, half> sum_result_store(reinterpret_cast<half*>(sum_result),
                                                           row);
  cuda::softmax::DispatchSoftmax<decltype(load), decltype(store), float>(
      ctx->hip_stream(), load, store, row, col, sub_result_store, sum_result_store);
}
}  // namespace

template<typename T, typename K>
class SparseSoftmaxCrossEntropyKernel final : public user_op::OpKernel {
 public:
  SparseSoftmaxCrossEntropyKernel() = default;
  ~SparseSoftmaxCrossEntropyKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* prediction = ctx->Tensor4ArgNameAndIndex("prediction", 0);
    const user_op::Tensor* label = ctx->Tensor4ArgNameAndIndex("label", 0);
    user_op::Tensor* prob = ctx->Tensor4ArgNameAndIndex("prob", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);

    const int64_t num_instances = label->shape().elem_cnt();
    CHECK_EQ(prediction->shape().elem_cnt() % num_instances, 0);
    const int64_t num_classes = prediction->shape().elem_cnt() / num_instances;
    const int64_t lower_bound = 0;
    const int64_t depth = ctx->Attr<int64_t>("depth");

    void* temp_storage = tmp_buffer->mut_dptr();
    const size_t reduce_temp_storage_bytes =
        GetReduceTempStorageSize<T>(num_instances, num_classes);
    const size_t temp_storage_bytes_offset = GetProbStorageSize<T>(num_instances, num_classes);
    T* sum_result = reinterpret_cast<T*>(reinterpret_cast<unsigned char*>(temp_storage)
                                         + reduce_temp_storage_bytes);
    T* sub_result = reinterpret_cast<T*>(reinterpret_cast<unsigned char*>(temp_storage)
                                         + reduce_temp_storage_bytes + temp_storage_bytes_offset);

    const K* labels = label->dptr<K>();
    T* y = out->mut_dptr<T>();
    ComputeProb<T>(ctx->device_ctx(), num_instances, num_classes, prediction->dptr<T>(),
                   prob->mut_dptr<T>(), sub_result, sum_result);

    ComputeResultGpu<T, K><<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0,
                             ctx->device_ctx()->hip_stream()>>>(
        num_instances, num_classes, depth, lower_bound, labels, sum_result, sub_result, y);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename K>
class SparseSoftmaxCrossEntropyKernel<float16, K> final : public user_op::OpKernel {
 public:
  SparseSoftmaxCrossEntropyKernel() = default;
  ~SparseSoftmaxCrossEntropyKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* prediction = ctx->Tensor4ArgNameAndIndex("prediction", 0);
    const user_op::Tensor* label = ctx->Tensor4ArgNameAndIndex("label", 0);
    user_op::Tensor* prob = ctx->Tensor4ArgNameAndIndex("prob", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);

    const int64_t num_instances = label->shape().elem_cnt();
    CHECK_EQ(prediction->shape().elem_cnt() % num_instances, 0);
    const int64_t num_classes = prediction->shape().elem_cnt() / num_instances;
    const int64_t lower_bound = 0;
    const int64_t depth = ctx->Attr<int64_t>("depth");

    void* temp_storage = tmp_buffer->mut_dptr();
    const size_t reduce_temp_storage_bytes =
        GetReduceTempStorageSize<float16>(num_instances, num_classes);
    const size_t temp_storage_bytes_offset =
        GetProbStorageSize<float16>(num_instances, num_classes);
    float16* sum_result = reinterpret_cast<float16*>(reinterpret_cast<unsigned char*>(temp_storage)
                                                     + reduce_temp_storage_bytes);
    float16* sub_result =
        reinterpret_cast<float16*>(reinterpret_cast<unsigned char*>(temp_storage)
                                   + reduce_temp_storage_bytes + temp_storage_bytes_offset);

    const K* labels = label->dptr<K>();
    float16* y = out->mut_dptr<float16>();
    ComputeProb<float16>(ctx->device_ctx(), num_instances, num_classes, prediction->dptr<float16>(),
                         prob->mut_dptr<float16>(), sub_result, sum_result);
    ComputeResultGpuHalf<K><<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0,
                              ctx->device_ctx()->hip_stream()>>>(
        num_instances, num_classes, depth, lower_bound, labels, reinterpret_cast<half*>(sum_result),
        reinterpret_cast<half*>(sub_result), reinterpret_cast<half*>(y));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_SPARSE_SOFTMAX_CROSS_ENTROPY_KERNEL(dtype_pair, ltype_pair)                     \
  REGISTER_USER_KERNEL("sparse_softmax_cross_entropy")                                           \
      .SetCreateFn<SparseSoftmaxCrossEntropyKernel<OF_PP_PAIR_FIRST(dtype_pair),                 \
                                                   OF_PP_PAIR_FIRST(ltype_pair)>>()              \
      .SetIsMatchedHob((user_op::HobDeviceTag() == DeviceType::kGPU)                             \
                       & (user_op::HobDataType("label", 0) == OF_PP_PAIR_SECOND(ltype_pair))     \
                       & (user_op::HobDataType("out", 0) == OF_PP_PAIR_SECOND(dtype_pair)))      \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                        \
        const Shape& prediction_shape = ctx->InputShape("prediction", 0);                        \
        const int64_t num_classes = prediction_shape.At(prediction_shape.NumAxes() - 1);         \
        const int64_t num_instances = prediction_shape.Count(0, prediction_shape.NumAxes() - 1); \
        return SparseSoftmaxCrossEntropyKernelUtil<                                              \
            DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype_pair),                                      \
            OF_PP_PAIR_FIRST(ltype_pair)>::GetComputeTempStorageSizeInBytes(num_instances,       \
                                                                            num_classes);        \
      });

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_SPARSE_SOFTMAX_CROSS_ENTROPY_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ FLOAT16_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

}  // namespace user_op
}  // namespace oneflow
