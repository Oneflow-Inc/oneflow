#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"

namespace oneflow {

namespace {

__global__ void DynamicLossScaleScheduleGpu(const int64_t increment_period, const float multiplier,
                                            const int64_t* count_not_finite, float* loss_scale,
                                            int64_t* good_step_counter) {
  if (*count_not_finite == 0) {
    int64_t cur_good_step_counter = *good_step_counter + 1;
    if (cur_good_step_counter >= increment_period) {
      *loss_scale = static_cast<float>(
          min(static_cast<double>(*loss_scale) * multiplier, static_cast<double>(FLT_MAX)));
      cur_good_step_counter = 0;
    }
    *good_step_counter = cur_good_step_counter;
  } else {
    *good_step_counter = 0;
    *loss_scale = static_cast<float>(max(static_cast<double>(*loss_scale) / multiplier, 1.0));
  }
}

}  // namespace

class DynamicLossScaleScheduleGpuKernel final : public user_op::OpKernel {
 public:
  DynamicLossScaleScheduleGpuKernel() = default;
  ~DynamicLossScaleScheduleGpuKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* count_not_finite = ctx->Tensor4ArgNameAndIndex("count_not_finite", 0);
    user_op::Tensor* loss_scale = ctx->Tensor4ArgNameAndIndex("loss_scale", 0);
    user_op::Tensor* good_step_counter = ctx->Tensor4ArgNameAndIndex("good_step_counter", 0);
    const auto increment_period = ctx->Attr<int64_t>("increment_period");
    const auto multiplier = ctx->Attr<float>("multiplier");
    DynamicLossScaleScheduleGpu<<<1, 1, 0, ctx->device_ctx()->hip_stream()>>>(
        increment_period, multiplier, count_not_finite->dptr<int64_t>(),
        loss_scale->mut_dptr<float>(), good_step_counter->mut_dptr<int64_t>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return true; }
};

REGISTER_USER_KERNEL("dynamic_loss_scale_schedule")
    .SetCreateFn<DynamicLossScaleScheduleGpuKernel>()
    .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu"));

}  // namespace oneflow
