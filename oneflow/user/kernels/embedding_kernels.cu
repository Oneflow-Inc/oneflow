#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/embedding/cuda_in_memory_key_value_store.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/ep/include/primitive/memcpy.h"
#include "oneflow/core/embedding/embedding_manager.h"

namespace oneflow {

namespace {

template<typename T, typename IDX>
__global__ void SGDUpdateKernel(const int64_t embedding_size, const IDX* num_unique_ids,
                                const float* learning_rate, float learning_rate_val,
                                const T* model_diff, const T* model, T* updated_model) {
  if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
  const int64_t n = *num_unique_ids * embedding_size;
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T model_val = model[i];
    updated_model[i] = model_val - learning_rate_val * model_diff[i];
  }
}

template<typename T, typename K, typename IDX>
__global__ void InitValueKernel(const int64_t embedding_size, const IDX* num_unique_ids,
                                const K* unique_ids, T* values) {
  const int64_t n = *num_unique_ids * embedding_size;
  CUDA_1D_KERNEL_LOOP(i, n) {
    IDX idx = i / embedding_size;
    values[i] = unique_ids[idx];
  }
}

class EmbeddingKernelState final : public user_op::OpKernelState {
 public:
  explicit EmbeddingKernelState(user_op::KernelInitContext* ctx) {
    OF_CUDA_CHECK(hipHostMalloc(&host_num_keys_, 1 * sizeof(int32_t)));  // TODO: int32_t->IDX
  }
  ~EmbeddingKernelState() { OF_CUDA_CHECK(hipHostFree(host_num_keys_)); }

  void* HostNumKeys() { return host_num_keys_; }

 private:
  void* host_num_keys_;
};

}  // namespace

template<typename K, typename IDX>
class EmbeddingPrefetchKernel final : public user_op::OpKernel {
 public:
  EmbeddingPrefetchKernel() = default;
  ~EmbeddingPrefetchKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingKernelState*>(state);
    CHECK(kernel_state != nullptr);
    LOG(ERROR) << "EmbeddingPrefetchKernel";
    embedding::KeyValueStore* store = Global<EmbeddingMgr>::Get()->GetKeyValueStore(
        "MyEmbeddingTest", ctx->parallel_ctx().parallel_id());
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    user_op::Tensor* context = ctx->Tensor4ArgNameAndIndex("context", 0);
    IDX* host_num_keys = reinterpret_cast<IDX*>(kernel_state->HostNumKeys());
    std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
        ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                  ep::primitive::MemcpyKind::kDtoH);
    CHECK(copyd2h_primitive);
    copyd2h_primitive->Launch(ctx->stream(), host_num_keys, num_unique_ids->dptr(), sizeof(IDX));
    CHECK_JUST(ctx->stream()->Sync());
    uint32_t num_keys = *host_num_keys;
    store->Prefetch(ctx->stream(), num_keys, unique_ids->dptr(),
                    reinterpret_cast<uint64_t*>(context->mut_dptr()));

    if (ParseBooleanFromEnv("DEBUG_SHUFFLE", false)) {
      int64_t embedding_size = 128;
      float* unique_values;
      OF_CUDA_CHECK(hipMalloc(&unique_values, num_keys * embedding_size * sizeof(float)));
      InitValueKernel<float, K, IDX>
          <<<BlocksNum4ThreadsNum(num_keys * embedding_size), kCudaThreadsNumPerBlock, 0,
             ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
              embedding_size, num_unique_ids->dptr<IDX>(), unique_ids->dptr<K>(), unique_values);
      store->Update(ctx->stream(), num_keys, unique_ids->dptr(),
                    reinterpret_cast<const uint64_t*>(context->dptr()), unique_values);
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_EMBEDDING_PREFETCH_KERNEL(k_dtype, idx_dtype) \
  REGISTER_USER_KERNEL("embedding_prefetch")                        \
      .SetCreateFn<EmbeddingPrefetchKernel<k_dtype, idx_dtype>>()   \
      .SetIsMatchedHob(                                             \
          (user_op::HobDeviceType() == DeviceType::kCUDA)           \
          && (user_op::HobDataType("num_unique_ids", 0) == GetDataType<idx_dtype>::value));

REGISTER_CUDA_EMBEDDING_PREFETCH_KERNEL(int64_t, int32_t)

template<typename T, typename K, typename IDX>
class EmbeddingLookupKernel final : public user_op::OpKernel {
 public:
  EmbeddingLookupKernel() = default;
  ~EmbeddingLookupKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingKernelState*>(state);
    CHECK(kernel_state != nullptr);
    embedding::KeyValueStore* store = Global<EmbeddingMgr>::Get()->GetKeyValueStore(
        "MyEmbeddingTest", ctx->parallel_ctx().parallel_id());

    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* context = ctx->Tensor4ArgNameAndIndex("context", 0);
    user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
    IDX* host_num_keys = reinterpret_cast<IDX*>(kernel_state->HostNumKeys());
    std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
        ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                  ep::primitive::MemcpyKind::kDtoH);
    CHECK(copyd2h_primitive);
    copyd2h_primitive->Launch(ctx->stream(), host_num_keys, num_unique_ids->dptr(), sizeof(IDX));
    CHECK_JUST(ctx->stream()->Sync());
    store->Lookup(ctx->stream(), *host_num_keys, unique_ids->dptr(),
                  reinterpret_cast<const uint64_t*>(context->dptr()), embeddings->mut_dptr());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL(t_dtype, k_dtype, idx_dtype)                \
  REGISTER_USER_KERNEL("embedding_lookup")                                                \
      .SetCreateFn<EmbeddingLookupKernel<t_dtype, k_dtype, idx_dtype>>()                  \
      .SetIsMatchedHob(                                                                   \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                 \
          && (user_op::HobDataType("num_unique_ids", 0) == GetDataType<idx_dtype>::value) \
          && (user_op::HobDataType("unique_ids", 0) == GetDataType<k_dtype>::value)       \
          && (user_op::HobDataType("embeddings", 0) == GetDataType<t_dtype>::value));

REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL(float, int64_t, int32_t)

template<typename T, typename K, typename IDX>
class EmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  EmbeddingUpdateKernel() = default;
  ~EmbeddingUpdateKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EmbeddingKernelState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* kernel_state = dynamic_cast<EmbeddingKernelState*>(state);
    CHECK(kernel_state != nullptr);
    LOG(ERROR) << "EmbeddingUpdateKernel";
    embedding::KeyValueStore* store = Global<EmbeddingMgr>::Get()->GetKeyValueStore(
        "MyEmbeddingTest", ctx->parallel_ctx().parallel_id());

    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* context = ctx->Tensor4ArgNameAndIndex("context", 0);
    const user_op::Tensor* unique_embeddings = ctx->Tensor4ArgNameAndIndex("unique_embeddings", 0);
    const user_op::Tensor* embedding_diff = ctx->Tensor4ArgNameAndIndex("embedding_diff", 0);
    const int64_t embedding_size =
        unique_embeddings->shape().elem_cnt() / unique_ids->shape().elem_cnt();
    LOG(ERROR) << "embedding_size " << embedding_size;
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    T* update_unique_embeddings = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>());

    IDX* host_num_keys = reinterpret_cast<IDX*>(kernel_state->HostNumKeys());
    std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
        ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                  ep::primitive::MemcpyKind::kDtoH);
    CHECK(copyd2h_primitive);
    copyd2h_primitive->Launch(ctx->stream(), host_num_keys, num_unique_ids->dptr(), sizeof(IDX));
    CHECK_JUST(ctx->stream()->Sync());

    const float learning_rate_val = ctx->Attr<float>("learning_rate_val");
    const float* learning_rate_ptr = nullptr;
    if (ctx->has_input("learning_rate", 0)) {
      const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
      learning_rate_ptr = learning_rate->dptr<float>();
    }
    // update kernel
    SGDUpdateKernel<T, IDX>
        <<<BlocksNum4ThreadsNum(embedding_diff->shape().elem_cnt()), kCudaThreadsNumPerBlock, 0,
           ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            embedding_size, num_unique_ids->dptr<IDX>(), learning_rate_ptr, learning_rate_val,
            embedding_diff->dptr<T>(), unique_embeddings->dptr<T>(), update_unique_embeddings);

    store->Update(ctx->stream(), *host_num_keys, unique_ids->dptr(),
                  reinterpret_cast<const uint64_t*>(context->dptr()), update_unique_embeddings);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_EMBEDDING_UPDATE_KERNEL(t_dtype, k_dtype, idx_dtype)                  \
  REGISTER_USER_KERNEL("sgd_embedding_update")                                              \
      .SetCreateFn<EmbeddingUpdateKernel<t_dtype, k_dtype, idx_dtype>>()                    \
      .SetIsMatchedHob(                                                                     \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                   \
          && (user_op::HobDataType("num_unique_ids", 0) == GetDataType<idx_dtype>::value)   \
          && (user_op::HobDataType("unique_ids", 0) == GetDataType<k_dtype>::value)         \
          && (user_op::HobDataType("unique_embeddings", 0) == GetDataType<t_dtype>::value)) \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                   \
        const user_op::TensorDesc& unique_embeddings =                                      \
            ctx->InputTensorDesc("unique_embeddings", 0);                                   \
        return unique_embeddings.shape().elem_cnt() * sizeof(t_dtype);                      \
      });

REGISTER_CUDA_EMBEDDING_UPDATE_KERNEL(float, int64_t, int32_t)

}  // namespace oneflow
