
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#ifdef WITH_CUTLASS

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/cuda/cuda_stream.h"
#include "oneflow/core/ep/include/primitive/permute.h"
#include "cutlass/arch/mma.h"
#include "cutlass/gemm/warp/mma.h"
#include "kernel_forward.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "trt_flash_attention/fmha.h"
#include "trt_flash_attention/fmha_flash_attention.h"

namespace oneflow {

namespace user_op {

namespace {

void ParseDims(const ShapeView& shape, const std::string& layout,
               const Optional<int64_t>& batch_size, const Optional<int64_t>& seq_len,
               const Optional<int64_t>& num_heads, const Optional<int64_t>& head_size,
               int64_t tensor_index, int64_t* b, int64_t* m, int64_t* h, int64_t* k,
               int64_t* b_stride, int64_t* m_stride, int64_t* h_stride, int64_t* offset,
               bool* bm_packed) {
  if (shape.NumAxes() == 2) {
    if (layout == "(BM)(HK)" || layout == "(BM)(H2K)" || layout == "(BM)(H3K)") {
      *bm_packed = true;
      CHECK(batch_size);
      CHECK(seq_len);
      *b = CHECK_JUST(batch_size);
      *m = CHECK_JUST(seq_len);
      int64_t packed_n = 0;
      if (layout == "(BM)(HK)") {
        packed_n = 1;
      } else if (layout == "(BM)(H2K)") {
        packed_n = 2;
      } else if (layout == "(BM)(H3K)") {
        packed_n = 3;
      } else {
        UNIMPLEMENTED();
      }
      const int64_t hidden_size = shape.At(1);
      if (num_heads) {
        const int64_t expected_h = CHECK_JUST(num_heads);
        const int64_t packed_h = packed_n * expected_h;
        CHECK_EQ(hidden_size % packed_h, 0);
        *h = expected_h;
        *k = hidden_size / packed_h;
      } else if (head_size) {
        const int64_t expected_k = CHECK_JUST(head_size);
        const int64_t packed_k = packed_n * expected_k;
        CHECK_EQ(hidden_size % packed_k, 0);
        *h = hidden_size / packed_k;
        *k = expected_k;
      } else {
        UNIMPLEMENTED();
      }
      *h_stride = *k * packed_n;
      *m_stride = *h_stride * *h;
      *b_stride = 0;
      if (packed_n == 1) {
        *offset = 0;
      } else if (packed_n == 2) {
        CHECK_GE(tensor_index, 1);
        *offset = (tensor_index - 1) * *k;
      } else if (packed_n == 3) {
        *offset = tensor_index * *k;
      } else {
        UNIMPLEMENTED();
      }
    } else {
      UNIMPLEMENTED();
    }
  } else if (shape.NumAxes() == 3) {
    if (layout == "BM(HK)" || layout == "BM(H2K)" || layout == "BM(H3K)" || layout == "MB(HK)"
        || layout == "MB(H2K)" || layout == "MB(H3K)") {
      *bm_packed = false;
      bool batch_first = false;
      int64_t packed_n = 0;
      const std::string layout_bm = layout.substr(0, 2);
      const std::string layout_hk = layout.substr(2);
      if (layout_bm == "BM") {
        *b = shape.At(0);
        *m = shape.At(1);
        batch_first = true;
      } else if (layout_bm == "MB") {
        *b = shape.At(1);
        *m = shape.At(0);
        batch_first = false;
      } else {
        UNIMPLEMENTED();
      }
      if (layout_hk == "(HK)") {
        packed_n = 1;
      } else if (layout_hk == "(H2K)") {
        packed_n = 2;
      } else if (layout_hk == "(H3K)") {
        packed_n = 3;
      } else {
        UNIMPLEMENTED();
      }
      const int64_t hidden_size = shape.At(2);
      if (num_heads) {
        const int64_t expected_h = CHECK_JUST(num_heads);
        const int64_t packed_h = packed_n * expected_h;
        CHECK_EQ(hidden_size % packed_h, 0);
        *h = expected_h;
        *k = hidden_size / packed_h;
      } else if (head_size) {
        const int64_t expected_k = CHECK_JUST(head_size);
        const int64_t packed_k = packed_n * expected_k;
        CHECK_EQ(hidden_size % packed_k, 0);
        *h = hidden_size / packed_k;
        *k = expected_k;
      } else {
        UNIMPLEMENTED();
      }
      *h_stride = *k * packed_n;
      if (batch_first) {
        *m_stride = *h_stride * *h;
        *b_stride = *m_stride * *m;
      } else {
        *b_stride = *h_stride * *h;
        *m_stride = *b_stride * *b;
      }
      if (packed_n == 1) {
        *offset = 0;
      } else if (packed_n == 2) {
        CHECK_GE(tensor_index, 1);
        *offset = (tensor_index - 1) * *k;
      } else if (packed_n == 3) {
        *offset = tensor_index * *k;
      } else {
        UNIMPLEMENTED();
      }
    } else if (layout == "(BM)HK") {
      *bm_packed = true;
      CHECK(batch_size);
      CHECK(seq_len);
      *b = CHECK_JUST(batch_size);
      *m = CHECK_JUST(seq_len);
      *h = shape.At(1);
      *k = shape.At(2);
      *h_stride = *k;
      *m_stride = *h_stride * *h;
      *b_stride = 0;
    } else {
      UNIMPLEMENTED();
    }
  } else if (shape.NumAxes() == 4) {
    *bm_packed = false;
    if (layout == "BMHK") {
      *b = shape.At(0);
      *m = shape.At(1);
      *h = shape.At(2);
      *k = shape.At(3);
      *h_stride = *k;
      *m_stride = *h_stride * *h;
      *b_stride = *m_stride * *m;
    } else if (layout == "BHMK") {
      *b = shape.At(0);
      *m = shape.At(2);
      *h = shape.At(1);
      *k = shape.At(3);
      *m_stride = *k;
      *h_stride = *m_stride * *m;
      *b_stride = *h_stride * *h;
    } else if (layout == "MBHK") {
      *b = shape.At(1);
      *m = shape.At(0);
      *h = shape.At(2);
      *k = shape.At(3);
      *h_stride = *k;
      *b_stride = *h_stride * *h;
      *m_stride = *b_stride * *b;
    } else {
      UNIMPLEMENTED();
    }
    *offset = 0;
  } else {
    UNIMPLEMENTED();
  };
  if (batch_size) {
    const int64_t expected_b = CHECK_JUST(batch_size);
    CHECK_EQ(*b, expected_b);
  }
  if (seq_len) {
    const int64_t expected_m = CHECK_JUST(seq_len);
    CHECK_EQ(*m, expected_m);
  }
  if (num_heads) {
    const int64_t expected_h = CHECK_JUST(num_heads);
    CHECK_EQ(*h, expected_h);
  }
  if (head_size) {
    const int64_t expected_k = CHECK_JUST(head_size);
    CHECK_EQ(*k, expected_k);
  }
}

void ParseDims(const ShapeView& shape, const std::string& layout,
               const Optional<int64_t>& num_heads, const Optional<int64_t>& head_size,
               int64_t tensor_index, int64_t* b, int64_t* m, int64_t* h, int64_t* k,
               int64_t* b_stride, int64_t* m_stride, int64_t* h_stride, int64_t* offset) {
  bool bm_packed{};
  ParseDims(shape, layout, Optional<int64_t>(), Optional<int64_t>(), num_heads, head_size,
            tensor_index, b, m, h, k, b_stride, m_stride, h_stride, offset, &bm_packed);
}

template<typename T, int pack_size>
struct alignas(pack_size * sizeof(T)) Pack {
  T elem[pack_size];
};

template<typename T>
__global__ void PackQkv(int b, int s, int nh, int d, const T* q, const T* k, const T* v, T* o,
                        int32_t* seq_len) {
  int count = b * s * nh * d * 3;
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < count; i += blockDim.x * gridDim.x) {
    int row = i / (d * 3);
    int out_col = i - row * (d * 3);
    T out;
    if (out_col < d) {
      out = q[row * d + out_col];
    } else if (out_col < 2 * d) {
      out = k[row * d + out_col - d];
    } else {
      out = v[row * d + out_col - d * 2];
    }
    o[i] = out;
  }
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < b + 1; i += blockDim.x * gridDim.x) {
    seq_len[i] = i * s;
  }
}

struct Params {
  DataType data_type;
  int64_t num_batches;
  int64_t num_heads;
  int64_t query_seq_len;
  int64_t kv_seq_len;
  int64_t head_size;
  int64_t value_head_size;
  int64_t q_stride_b;
  int64_t q_stride_m;
  int64_t q_stride_h;
  int64_t k_stride_b;
  int64_t k_stride_m;
  int64_t k_stride_h;
  int64_t v_stride_b;
  int64_t v_stride_m;
  int64_t v_stride_h;
  std::string attn_mask_type;
  int64_t causal_diagonal_offset;
  const void* query_ptr;
  const void* key_ptr;
  const void* value_ptr;
  const void* attn_bias_ptr;
  const void* query_seq_start_ptr;
  const void* key_seq_start_ptr;
  const void* key_seq_len_ptr;
  int64_t attn_bias_stride_b;
  int64_t attn_bias_stride_h;
  int64_t attn_bias_stride_m;
  void* out_ptr;
  void* workspace;
  int64_t workspace_size;
  float scale;
};

template<typename T, typename ArchTag, bool is_aligned, int queries_per_block, int keys_per_block,
         bool single_value_iteration, bool with_attn_bias>
void LaunchCutlassFmha(const Params& params, ep::CudaStream* stream) {
  // The fmha implementation below is based on xformers's fmha
  // implementation at:
  // https://github.com/facebookresearch/xformers/tree/main/xformers/csrc/attention/cuda/fmha
  using Attention = AttentionKernel<T, ArchTag, is_aligned, queries_per_block, keys_per_block,
                                    single_value_iteration, false, with_attn_bias>;
  typename Attention::Params p{};
  p.query_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.query_ptr));
  p.key_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.key_ptr));
  p.value_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.value_ptr));
  p.attn_bias_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.attn_bias_ptr));
  p.seqstart_q_ptr =
      const_cast<int32_t*>(reinterpret_cast<const int32_t*>(params.query_seq_start_ptr));
  p.seqstart_k_ptr =
      const_cast<int32_t*>(reinterpret_cast<const int32_t*>(params.key_seq_start_ptr));
  p.seqlen_k_ptr = const_cast<int32_t*>(reinterpret_cast<const int32_t*>(params.key_seq_len_ptr));
  p.logsumexp_ptr = nullptr;
  p.output_ptr = reinterpret_cast<T*>(params.out_ptr);
  if (Attention::kNeedsOutputAccumulatorBuffer) {
    using Acc = typename Attention::accum_t;
    CHECK_GE(params.workspace_size, params.num_batches * params.query_seq_len * params.num_heads
                                        * params.value_head_size * sizeof(Acc));
    p.output_accum_ptr = reinterpret_cast<Acc*>(params.workspace);
  } else {
    p.output_accum_ptr = nullptr;
  }
  p.num_heads = params.num_heads;
  p.num_batches = params.num_batches;
  p.head_dim = params.head_size;
  p.head_dim_value = params.value_head_size;
  p.num_queries = params.query_seq_len;
  p.num_keys = params.kv_seq_len;
  p.q_strideM = params.q_stride_m;
  p.k_strideM = params.k_stride_m;
  p.v_strideM = params.v_stride_m;
  p.o_strideM = p.head_dim_value * p.num_heads;
  p.bias_strideM = params.attn_bias_stride_m;

  p.q_strideH = params.q_stride_h;
  p.k_strideH = params.k_stride_h;
  p.v_strideH = params.v_stride_h;
  p.bias_strideH = params.attn_bias_stride_h;

  p.q_strideB = params.q_stride_b;
  p.k_strideB = params.k_stride_b;
  p.v_strideB = params.v_stride_b;
  p.bias_strideB = params.attn_bias_stride_b;

  p.scale = params.scale;

  if (params.attn_mask_type == "none") {
    p.custom_mask_type = Attention::NoCustomMask;
  } else if (params.attn_mask_type == "causal_from_top_left") {
    p.custom_mask_type = Attention::CausalFromTopLeft;
  } else if (params.attn_mask_type == "causal_from_bottom_right") {
    p.custom_mask_type = Attention::CausalFromBottomRight;
  } else {
    UNIMPLEMENTED();
  }
  p.causal_diagonal_offset = params.causal_diagonal_offset;
  p.use_dropout = false;

  constexpr auto kernel_fn = attention_kernel_batched_impl<Attention>;
  int smem_bytes = sizeof(typename Attention::SharedStorage);
  if (smem_bytes > 0xc000) {
    static bool once = [&]() {
      cudaFuncSetAttribute(kernel_fn, cudaFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
      return true;
    }();
  }
  CHECK(Attention::check_supported(p));
  kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes, stream->cuda_stream()>>>(p);
}

template<typename T, typename ArchTag, bool is_aligned, int queries_per_block, int keys_per_block,
         bool single_value_iteration>
void DispatchWithAttnBias(const Params& params, ep::CudaStream* stream) {
  if (params.attn_bias_ptr != nullptr) {
    LaunchCutlassFmha<T, ArchTag, is_aligned, queries_per_block, keys_per_block,
                      single_value_iteration, true>(params, stream);
  } else {
    LaunchCutlassFmha<T, ArchTag, is_aligned, queries_per_block, keys_per_block,
                      single_value_iteration, false>(params, stream);
  }
}

template<typename T, typename ArchTag, bool is_aligned, int queries_per_block, int keys_per_block>
void DispatchSingleValueIteration(const Params& params, ep::CudaStream* stream) {
  if (params.value_head_size <= keys_per_block) {
    DispatchWithAttnBias<T, ArchTag, is_aligned, queries_per_block, keys_per_block, true>(params,
                                                                                          stream);
  } else {
    DispatchWithAttnBias<T, ArchTag, is_aligned, queries_per_block, keys_per_block, false>(params,
                                                                                           stream);
  }
}

template<typename T, typename ArchTag, bool is_aligned>
void DispatchKeysPerBlock(const Params& params, ep::CudaStream* stream) {
  if (params.value_head_size <= 64) {
    DispatchSingleValueIteration<T, ArchTag, is_aligned, 64, 64>(params, stream);
  } else {
    DispatchSingleValueIteration<T, ArchTag, is_aligned, 32, 128>(params, stream);
  }
}

template<typename T, typename ArchTag>
void DispatchIsAligned(const Params& params, ep::CudaStream* stream) {
  if (reinterpret_cast<uintptr_t>(params.query_ptr) % 16 == 0
      && reinterpret_cast<uintptr_t>(params.key_ptr) % 16 == 0
      && reinterpret_cast<uintptr_t>(params.value_ptr) % 16 == 0
      && params.attn_bias_stride_m % (16 / sizeof(T)) == 0
      && params.head_size % (16 / sizeof(T)) == 0
      && params.value_head_size % (16 / sizeof(T)) == 0) {
    DispatchKeysPerBlock<T, ArchTag, true>(params, stream);
  } else {
    DispatchKeysPerBlock<T, ArchTag, false>(params, stream);
  }
}

template<typename T>
void DispatchArchTag(const Params& params, ep::CudaStream* stream) {
  const int major = stream->device_properties().major;
  const int minor = stream->device_properties().minor;

  if (major == 8) {
    DispatchIsAligned<T, cutlass::arch::Sm80>(params, stream);
  } else if (major == 7) {
    if (minor == 5) {
      DispatchIsAligned<T, cutlass::arch::Sm75>(params, stream);
    } else {
      DispatchIsAligned<T, cutlass::arch::Sm70>(params, stream);
    }
  } else {
    UNIMPLEMENTED();
  }
}

void DispatchCutlassFmha(const Params& params, ep::CudaStream* stream) {
  if (params.data_type == DataType::kFloat16) {
    DispatchArchTag<cutlass::half_t>(params, stream);
  } else if (params.data_type == DataType::kFloat) {
    DispatchArchTag<float>(params, stream);
  } else {
    UNIMPLEMENTED();
  }
}

class FusedMultiHeadAttentionInferenceKernel final : public user_op::OpKernel,
                                                     public user_op::CudaGraphSupport {
 public:
  FusedMultiHeadAttentionInferenceKernel() = default;
  ~FusedMultiHeadAttentionInferenceKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const Tensor* query = ctx->Tensor4ArgNameAndIndex("query", 0);
    const Tensor* key = ctx->Tensor4ArgNameAndIndex("key", 0);
    const Tensor* value = ctx->Tensor4ArgNameAndIndex("value", 0);
    const Tensor* attn_bias = nullptr;
    if (ctx->has_input("attn_bias", 0)) { attn_bias = ctx->Tensor4ArgNameAndIndex("attn_bias", 0); }
    const Tensor* query_seq_start = nullptr;
    const Tensor* key_seq_start = nullptr;
    const Tensor* key_seq_len = nullptr;
    const float scale = ctx->Attr<double>("scale");
    if (ctx->has_input("query_seq_start", 0)) {
      CHECK(ctx->has_input("key_seq_start", 0));
      query_seq_start = ctx->Tensor4ArgNameAndIndex("query_seq_start", 0);
      key_seq_start = ctx->Tensor4ArgNameAndIndex("key_seq_start", 0);
      CHECK(query_seq_start->data_type() == DataType::kInt32);
      CHECK(key_seq_start->data_type() == DataType::kInt32);
      CHECK_EQ(query_seq_start->shape_view().NumAxes(), 1);
      CHECK_GT(query_seq_start->shape_view().At(0), 1);
      CHECK(query_seq_start->shape_view() == key_seq_start->shape_view());
      if (ctx->has_input("key_seq_len", 0)) {
        key_seq_len = ctx->Tensor4ArgNameAndIndex("key_seq_len", 0);
        CHECK(key_seq_len->data_type() == DataType::kInt32);
        CHECK_EQ(key_seq_len->shape_view().NumAxes(), 1);
        CHECK_EQ(key_seq_len->shape_view().At(0), query_seq_start->shape_view().At(0) - 1);
      }
    } else {
      CHECK(!ctx->has_input("key_seq_start", 0));
      CHECK(!ctx->has_input("key_seq_len", 0));
    }
    Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    Tensor* tmp = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const DataType data_type = query->data_type();
    CHECK_EQ(key->data_type(), data_type);
    CHECK_EQ(value->data_type(), data_type);
    CHECK_EQ(out->data_type(), data_type);
    const int64_t query_head_size = ctx->Attr<int64_t>("query_head_size");
    const std::string& attn_mask_type = ctx->Attr<std::string>("attn_mask_type");
    const int64_t causal_diagonal_offset = ctx->Attr<int64_t>("causal_diagonal_offset");
    CHECK_GE(causal_diagonal_offset, 0);
    const std::string& query_layout = ctx->Attr<std::string>("query_layout");
    const std::string& key_layout = ctx->Attr<std::string>("key_layout");
    const std::string& value_layout = ctx->Attr<std::string>("value_layout");
    const std::string& output_layout = ctx->Attr<std::string>("output_layout");

    Optional<int64_t> batch_size;
    if (query_seq_start != nullptr) { batch_size = query_seq_start->shape_view().At(0) - 1; }
    Optional<int64_t> query_max_seq_len;
    const int64_t attr_query_max_seq_len = ctx->Attr<int64_t>("query_max_seq_len");
    if (attr_query_max_seq_len != 0) { query_max_seq_len = attr_query_max_seq_len; }
    Optional<int64_t> key_max_seq_len;
    const int64_t attr_key_max_seq_len = ctx->Attr<int64_t>("key_max_seq_len");
    if (attr_key_max_seq_len != 0) { key_max_seq_len = attr_key_max_seq_len; }

    int64_t q_b = 0;
    int64_t q_m = 0;
    int64_t q_h = 0;
    int64_t q_k = 0;
    int64_t q_b_stride = 0;
    int64_t q_m_stride = 0;
    int64_t q_h_stride = 0;
    int64_t q_offset = 0;
    bool q_bm_packed = false;
    ParseDims(query->shape_view(), query_layout, batch_size, query_max_seq_len, Optional<int64_t>(),
              query_head_size, 0, &q_b, &q_m, &q_h, &q_k, &q_b_stride, &q_m_stride, &q_h_stride,
              &q_offset, &q_bm_packed);
    if (q_bm_packed) { CHECK(query_seq_start != nullptr); }

    int64_t k_b = 0;
    int64_t k_m = 0;
    int64_t k_h = 0;
    int64_t k_k = 0;
    int64_t k_b_stride = 0;
    int64_t k_m_stride = 0;
    int64_t k_h_stride = 0;
    int64_t k_offset = 0;
    bool k_bm_packed = false;
    ParseDims(key->shape_view(), key_layout, q_b, key_max_seq_len, Optional<int64_t>(),
              query_head_size, 1, &k_b, &k_m, &k_h, &k_k, &k_b_stride, &k_m_stride, &k_h_stride,
              &k_offset, &k_bm_packed);
    CHECK_EQ(k_b, q_b);
    CHECK_EQ(k_h, q_h);
    CHECK_EQ(k_bm_packed, q_bm_packed);

    int64_t v_b = 0;
    int64_t v_m = 0;
    int64_t v_h = 0;
    int64_t v_k = 0;
    int64_t v_b_stride = 0;
    int64_t v_m_stride = 0;
    int64_t v_h_stride = 0;
    int64_t v_offset = 0;
    bool v_bm_packed = false;
    ParseDims(value->shape_view(), value_layout, q_b, k_m, q_h, Optional<int64_t>(), 2, &v_b, &v_m,
              &v_h, &v_k, &v_b_stride, &v_m_stride, &v_h_stride, &v_offset, &v_bm_packed);
    CHECK_EQ(v_b, q_b);
    CHECK_EQ(v_m, k_m);
    CHECK_EQ(v_bm_packed, k_bm_packed);
    if (output_layout == "BM(HK)") {
      CHECK(!q_bm_packed);
      CHECK_EQ(out->shape_view().NumAxes(), 3);
      CHECK_EQ(out->shape_view().At(0), q_b);
      CHECK_EQ(out->shape_view().At(1), q_m);
      CHECK_EQ(out->shape_view().At(2), q_h * v_k);
    } else if (output_layout == "MB(HK)") {
      CHECK(!q_bm_packed);
      CHECK_EQ(out->shape_view().NumAxes(), 3);
      CHECK_EQ(q_b, 1);
      CHECK_EQ(out->shape_view().At(0), q_m);
      CHECK_EQ(out->shape_view().At(1), q_b);
      CHECK_EQ(out->shape_view().At(2), q_h * v_k);
    } else if (output_layout == "(BM)(HK)") {
      CHECK(q_bm_packed);
      CHECK_EQ(out->shape_view().NumAxes(), 2);
      CHECK_EQ(out->shape_view().At(0), query->shape_view().At(0));
      CHECK_EQ(out->shape_view().At(1), q_h * v_k);
    } else {
      UNIMPLEMENTED();
    }

    auto* cuda_stream = ctx->stream()->As<ep::CudaStream>();

    // Compatible with typo `KERENL`
    const bool enable_trt_flash_attn =
        ParseBooleanFromEnv(
            "ONEFLOW_KERNEL_FMHA_ENABLE_TRT_FLASH_ATTN_IMPL",
            ParseBooleanFromEnv("ONEFLOW_KERENL_FMHA_ENABLE_TRT_FLASH_ATTN_IMPL", true))
        && ParseBooleanFromEnv("ONEFLOW_MATMUL_ALLOW_HALF_PRECISION_ACCUMULATION", false);
    const bool is_default_scale =
        std::abs(scale - 1.0 / std::sqrt(static_cast<float>(q_k))) <= 1e-5;
    const int arch = cuda_stream->cuda_arch() / 10;
    const bool is_trt_supported_arch = (arch == 75 || arch == 80 || arch == 86 || arch == 89);
    const bool is_trt_supported_head_size = ((q_k == 40) || (q_k == 64));
    // Avoid PackQKV overhead when seq_len is small.
    const bool is_long_seq_len = q_m >= 512;
    const bool is_trt_supported_layout = (query_layout == "BMHK" || query_layout == "BM(HK)")
                                         && (key_layout == "BMHK" || key_layout == "BM(HK)")
                                         && (value_layout == "BMHK" || value_layout == "BM(HK)")
                                         && (output_layout == "BMHK" || output_layout == "BM(HK)");
    if (is_default_scale && query_seq_start == nullptr && enable_trt_flash_attn
        && data_type == DataType::kFloat16 && q_m == k_m && q_k == v_k && is_trt_supported_head_size
        && is_long_seq_len && is_trt_supported_arch && attn_mask_type == "none"
        && attn_bias == nullptr && is_trt_supported_layout) {
      // The fmha implementation below is based on TensorRT's multiHeadFlashAttentionPlugin
      // implementation at:
      // https://github.com/NVIDIA/TensorRT/tree/main/plugin/multiHeadFlashAttentionPlugin
      int32_t cu_seqlens_d_size = (q_b + 1) * sizeof(int32_t);
      int32_t* cu_seqlens_d = reinterpret_cast<int32_t*>(tmp->mut_dptr());
      half* packed_qkv =
          reinterpret_cast<half*>(tmp->mut_dptr<char>() + GetCudaAlignedSize(cu_seqlens_d_size));
      constexpr int pack_size = 4;
      using PackType = Pack<half, pack_size>;
      const int64_t count = q_b * q_m * q_h * q_k * 3 / pack_size;
      PackQkv<PackType><<<(count - 1 + 256) / 256, 256, 0, cuda_stream->cuda_stream()>>>(
          q_b, q_m, q_h, q_k / pack_size, reinterpret_cast<const PackType*>(query->dptr()),
          reinterpret_cast<const PackType*>(key->dptr()),
          reinterpret_cast<const PackType*>(value->dptr()), reinterpret_cast<PackType*>(packed_qkv),
          cu_seqlens_d);

#ifdef WITH_CUDA_GRAPHS
      cudaStreamCaptureMode mode = cudaStreamCaptureModeRelaxed;
      if (cuda_stream->IsGraphCapturing()) {
        OF_CUDA_CHECK(cudaThreadExchangeStreamCaptureMode(&mode));
      }
#endif  // WITH_CUDA_GRAPHS
      nvinfer1::plugin::FusedMultiHeadFlashAttentionKernel const* kernels =
          nvinfer1::plugin::getFMHAFlashCubinKernels(nvinfer1::plugin::DATA_TYPE_FP16, arch);
#ifdef WITH_CUDA_GRAPHS
      if (cuda_stream->IsGraphCapturing()) {
        OF_CUDA_CHECK(cudaThreadExchangeStreamCaptureMode(&mode));
      }
#endif  // WITH_CUDA_GRAPHS
      nvinfer1::plugin::runFMHFAKernel(packed_qkv, cu_seqlens_d, out->mut_dptr(), q_b * q_m, arch,
                                       kernels, q_b, q_h, q_k, q_m, cuda_stream->cuda_stream());
      return;
    }

    Params params{};
    params.data_type = data_type;
    params.num_batches = q_b;
    params.num_heads = q_h;
    params.query_seq_len = q_m;
    params.kv_seq_len = k_m;
    params.head_size = q_k;
    params.value_head_size = v_k;
    params.scale = scale;
    params.q_stride_b = q_b_stride;
    params.q_stride_m = q_m_stride;
    params.q_stride_h = q_h_stride;
    params.k_stride_b = k_b_stride;
    params.k_stride_m = k_m_stride;
    params.k_stride_h = k_h_stride;
    params.v_stride_b = v_b_stride;
    params.v_stride_m = v_m_stride;
    params.v_stride_h = v_h_stride;
    params.query_ptr = query->dptr<char>() + q_offset * GetSizeOfDataType(data_type);
    params.key_ptr = key->dptr<char>() + k_offset * GetSizeOfDataType(data_type);
    params.value_ptr = value->dptr<char>() + v_offset * GetSizeOfDataType(data_type);
    params.query_seq_start_ptr =
        query_seq_start == nullptr ? nullptr : query_seq_start->dptr<int32_t>();
    params.key_seq_start_ptr = key_seq_start == nullptr ? nullptr : key_seq_start->dptr<int32_t>();
    params.key_seq_len_ptr = key_seq_len == nullptr ? nullptr : key_seq_len->dptr<int32_t>();
    params.out_ptr = out->mut_dptr();
    const int64_t tmp_buffer_size = tmp->shape_view().elem_cnt();
    params.workspace = tmp->mut_dptr();
    params.workspace_size = tmp_buffer_size;
    params.attn_mask_type = attn_mask_type;
    params.causal_diagonal_offset = causal_diagonal_offset;
    if (attn_bias != nullptr) {
      const int64_t num_attn_bias_axes = attn_bias->shape_view().NumAxes();
      CHECK_GE(num_attn_bias_axes, 1);
      CHECK_LE(num_attn_bias_axes, 4);
      DimVector padded_attn_bias_shape;
      for (int i = 0; i < 4 - num_attn_bias_axes; ++i) { padded_attn_bias_shape.push_back(1); }
      for (int i = 0; i < num_attn_bias_axes; ++i) {
        padded_attn_bias_shape.push_back(attn_bias->shape_view().At(i));
      }
      CHECK_GE(padded_attn_bias_shape.at(3), k_m);
      int64_t bias_stride = padded_attn_bias_shape.at(3);
      if (padded_attn_bias_shape.at(2) == 1) {
        params.attn_bias_stride_m = 0;
      } else {
        CHECK_GE(padded_attn_bias_shape.at(2), q_m);
        params.attn_bias_stride_m = bias_stride;
        bias_stride *= padded_attn_bias_shape.at(2);
      }
      if (padded_attn_bias_shape.at(1) == 1) {
        params.attn_bias_stride_h = 0;
      } else {
        CHECK_EQ(padded_attn_bias_shape.at(1), q_h);
        params.attn_bias_stride_h = bias_stride;
        bias_stride *= q_h;
      }
      if (padded_attn_bias_shape.at(0) == 1) {
        params.attn_bias_stride_b = 0;
      } else {
        CHECK_EQ(padded_attn_bias_shape.at(0), q_b);
        params.attn_bias_stride_b = bias_stride;
      }
      params.attn_bias_ptr = attn_bias->dptr();
    } else {
      params.attn_bias_ptr = nullptr;
      params.attn_bias_stride_m = 0;
      params.attn_bias_stride_h = 0;
      params.attn_bias_stride_b = 0;
    }
    DispatchCutlassFmha(params, cuda_stream);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

size_t InferTmpBufferSize(InferContext* ctx) {
  const auto& out_desc = ctx->OutputTensorDesc("out", 0);
  size_t buffer_size = 0;
  buffer_size +=
      GetCudaAlignedSize(out_desc.shape().elem_cnt() * GetSizeOfDataType(DataType::kFloat));
  buffer_size +=
      GetCudaAlignedSize(out_desc.shape().elem_cnt() * GetSizeOfDataType(out_desc.data_type())) * 3;
  buffer_size +=
      GetCudaAlignedSize((out_desc.shape().At(0) + 1) * GetSizeOfDataType(DataType::kInt32));
  return buffer_size;
}

#define REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(dtype)    \
  REGISTER_USER_KERNEL("fused_multi_head_attention_inference")         \
      .SetCreateFn<FusedMultiHeadAttentionInferenceKernel>()           \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("out", 0) == dtype))   \
      .SetInferTmpSizeFn(InferTmpBufferSize);

REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(DataType::kFloat16)
REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(DataType::kFloat)

template<typename Index>
struct ConcatParam {
  const void* past_ptr;
  const void* ptr;
  void* output_ptr;
  Index past_offset;
  Index offset;
  Index output_offset;
  Index past_m;
  Index past_stride_b;
  Index past_stride_m;
  Index past_stride_h;
  Index stride_b;
  Index stride_m;
  Index stride_h;
  Index output_stride_b;
  Index output_stride_m;
  Index output_stride_h;
  Index count;
  Index output_khm;
  Index output_kh;
  Index output_k;
};

template<typename Index>
struct BatchConcatParam {
  ConcatParam<Index> params[2];
};

template<typename T, typename Index>
__device__ void ConcatPastKeyValue(ConcatParam<Index> p) {
  for (Index i = blockIdx.x * blockDim.x + threadIdx.x; i < p.count; i += blockDim.x * gridDim.x) {
    Index b_idx = i / p.output_khm;
    Index b_off = i - b_idx * p.output_khm;
    Index m_idx = b_off / p.output_kh;
    Index m_off = b_off - m_idx * p.output_kh;
    Index h_idx = m_off / p.output_k;
    Index k_idx = m_off - h_idx * p.output_k;
    T v;
    if (m_idx < p.past_m) {
      v = reinterpret_cast<const T*>(
          p.past_ptr)[p.past_offset + b_idx * p.past_stride_b + m_idx * p.past_stride_m
                      + h_idx * p.past_stride_h + k_idx];
    } else {
      v = reinterpret_cast<const T*>(
          p.ptr)[p.offset + b_idx * p.stride_b + (m_idx - p.past_m) * p.stride_m
                 + h_idx * p.stride_h + k_idx];
    }
    reinterpret_cast<T*>(
        p.output_ptr)[p.output_offset + b_idx * p.output_stride_b + m_idx * p.output_stride_m
                      + h_idx * p.output_stride_h + k_idx] = v;
  }
}

template<size_t elem_size, typename Index>
__global__ void BatchConcatPastKeyValue(BatchConcatParam<Index> params) {
  if (blockIdx.y == 0) {
    ConcatPastKeyValue<std::aligned_storage<elem_size, elem_size>::type, Index>(params.params[0]);
  } else if (blockIdx.y == 1) {
    ConcatPastKeyValue<std::aligned_storage<elem_size, elem_size>::type, Index>(params.params[1]);
  } else {
    // do nothing
  }
}

class FusedAttentionConcatPastKeyValueKernel final : public user_op::OpKernel,
                                                     public user_op::CudaGraphSupport {
 public:
  FusedAttentionConcatPastKeyValueKernel() = default;
  ~FusedAttentionConcatPastKeyValueKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const Tensor* key = ctx->Tensor4ArgNameAndIndex("key", 0);
    const Tensor* value = ctx->Tensor4ArgNameAndIndex("value", 0);
    Tensor* output_key = ctx->Tensor4ArgNameAndIndex("output_key", 0);
    Tensor* output_value = ctx->Tensor4ArgNameAndIndex("output_value", 0);
    const DataType data_type = key->data_type();
    const Tensor* past_key = nullptr;
    const Tensor* past_value = nullptr;
    if (ctx->has_input("past_key", 0)) {
      CHECK(ctx->has_input("past_value", 0));
      past_key = ctx->Tensor4ArgNameAndIndex("past_key", 0);
      past_value = ctx->Tensor4ArgNameAndIndex("past_value", 0);
      CHECK_EQ(past_key->data_type(), data_type);
      CHECK_EQ(past_value->data_type(), data_type);
    } else {
      CHECK(!ctx->has_input("past_value", 0));
    }
    CHECK_EQ(value->data_type(), data_type);
    CHECK_EQ(output_key->data_type(), data_type);
    CHECK_EQ(output_value->data_type(), data_type);
    const int64_t size_of_data_type = GetSizeOfDataType(data_type);
    const int64_t key_head_size = ctx->Attr<int64_t>("key_head_size");
    const std::string& past_key_layout = ctx->Attr<std::string>("past_key_layout");
    const std::string& past_value_layout = ctx->Attr<std::string>("past_value_layout");
    const std::string& key_layout = ctx->Attr<std::string>("key_layout");
    const std::string& value_layout = ctx->Attr<std::string>("value_layout");

    int64_t pack_size = 16 / size_of_data_type;
    while (key_head_size % pack_size != 0) { pack_size /= 2; }

    auto ParsePackedDims =
        [](const ShapeView& shape, const std::string& layout, const Optional<int64_t>& num_heads,
           const Optional<int64_t>& head_size, int64_t tensor_index, int64_t* b, int64_t* m,
           int64_t* h, int64_t* k, int64_t* b_stride, int64_t* m_stride, int64_t* h_stride,
           int64_t* offset, int64_t pack_size) {
          ParseDims(shape, layout, num_heads, head_size, tensor_index, b, m, h, k, b_stride,
                    m_stride, h_stride, offset);
          *k /= pack_size;
          *b_stride /= pack_size;
          *m_stride /= pack_size;
          *h_stride /= pack_size;
          *offset /= pack_size;
        };

    int64_t key_b = 0;
    int64_t key_m = 0;
    int64_t key_h = 0;
    int64_t key_k = 0;
    int64_t key_b_stride = 0;
    int64_t key_m_stride = 0;
    int64_t key_h_stride = 0;
    int64_t key_offset = 0;
    ParsePackedDims(key->shape_view(), key_layout, Optional<int64_t>(), key_head_size, 1, &key_b,
                    &key_m, &key_h, &key_k, &key_b_stride, &key_m_stride, &key_h_stride,
                    &key_offset, pack_size);

    int64_t value_b = 0;
    int64_t value_m = 0;
    int64_t value_h = 0;
    int64_t value_k = 0;
    int64_t value_b_stride = 0;
    int64_t value_m_stride = 0;
    int64_t value_h_stride = 0;
    int64_t value_offset = 0;
    ParsePackedDims(value->shape_view(), value_layout, key_h, key_head_size, 2, &value_b, &value_m,
                    &value_h, &value_k, &value_b_stride, &value_m_stride, &value_h_stride,
                    &value_offset, pack_size);
    CHECK_EQ(value_b, key_b);
    CHECK_EQ(value_m, key_m);

    int64_t past_key_b = 0;
    int64_t past_key_m = 0;
    int64_t past_key_h = 0;
    int64_t past_key_k = 0;
    int64_t past_key_b_stride = 0;
    int64_t past_key_m_stride = 0;
    int64_t past_key_h_stride = 0;
    int64_t past_key_offset = 0;
    if (past_key != nullptr) {
      ParsePackedDims(past_key->shape_view(), past_key_layout, key_h, key_head_size, 1, &past_key_b,
                      &past_key_m, &past_key_h, &past_key_k, &past_key_b_stride, &past_key_m_stride,
                      &past_key_h_stride, &past_key_offset, pack_size);
    }

    int64_t past_value_b = 0;
    int64_t past_value_m = 0;
    int64_t past_value_h = 0;
    int64_t past_value_k = 0;
    int64_t past_value_b_stride = 0;
    int64_t past_value_m_stride = 0;
    int64_t past_value_h_stride = 0;
    int64_t past_value_offset = 0;
    if (past_value != nullptr) {
      ParsePackedDims(past_value->shape_view(), past_value_layout, key_h, key_head_size, 2,
                      &past_value_b, &past_value_m, &past_value_h, &past_value_k,
                      &past_value_b_stride, &past_value_m_stride, &past_value_h_stride,
                      &past_value_offset, pack_size);
    }
    CHECK_EQ(past_value_b, past_key_b);
    CHECK_EQ(past_value_m, past_key_m);

    int64_t output_key_b = 0;
    int64_t output_key_m = 0;
    int64_t output_key_h = 0;
    int64_t output_key_k = 0;
    int64_t output_key_b_stride = 0;
    int64_t output_key_m_stride = 0;
    int64_t output_key_h_stride = 0;
    int64_t output_key_offset = 0;
    ParsePackedDims(output_key->shape_view(), past_key_layout, key_h, key_head_size, 1,
                    &output_key_b, &output_key_m, &output_key_h, &output_key_k,
                    &output_key_b_stride, &output_key_m_stride, &output_key_h_stride,
                    &output_key_offset, pack_size);
    CHECK_EQ(output_key_b, key_b);
    CHECK_EQ(output_key_m, past_key_m + key_m);

    int64_t output_value_b = 0;
    int64_t output_value_m = 0;
    int64_t output_value_h = 0;
    int64_t output_value_k = 0;
    int64_t output_value_b_stride = 0;
    int64_t output_value_m_stride = 0;
    int64_t output_value_h_stride = 0;
    int64_t output_value_offset = 0;
    ParsePackedDims(output_value->shape_view(), past_value_layout, key_h, key_head_size, 2,
                    &output_value_b, &output_value_m, &output_value_h, &output_value_k,
                    &output_value_b_stride, &output_value_m_stride, &output_value_h_stride,
                    &output_value_offset, pack_size);
    CHECK_EQ(output_value_b, key_b);
    CHECK_EQ(output_value_m, past_value_m + value_m);

    int64_t max_tensor_elem = (1 << 30) * pack_size;
    CHECK((past_key == nullptr || past_key->shape_view().elem_cnt() <= max_tensor_elem)
          && (past_value == nullptr || past_value->shape_view().elem_cnt() <= max_tensor_elem)
          && key->shape_view().elem_cnt() <= max_tensor_elem
          && value->shape_view().elem_cnt() <= max_tensor_elem
          && output_key->shape_view().elem_cnt() <= max_tensor_elem
          && output_value->shape_view().elem_cnt() <= max_tensor_elem);

    int64_t count = output_key_b * output_key_m * output_key_h * output_key_k;
    BatchConcatParam<int32_t> kv;

    kv.params[0].past_ptr = past_key == nullptr ? nullptr : past_key->dptr();
    kv.params[0].ptr = key->dptr();
    kv.params[0].output_ptr = output_key->mut_dptr();
    kv.params[0].past_offset = past_key_offset;
    kv.params[0].offset = key_offset;
    kv.params[0].output_offset = output_key_offset;
    kv.params[0].past_m = past_key_m;
    kv.params[0].past_stride_b = past_key_b_stride;
    kv.params[0].past_stride_m = past_key_m_stride;
    kv.params[0].past_stride_h = past_key_h_stride;
    kv.params[0].stride_b = key_b_stride;
    kv.params[0].stride_m = key_m_stride;
    kv.params[0].stride_h = key_h_stride;
    kv.params[0].output_stride_b = output_key_b_stride;
    kv.params[0].output_stride_m = output_key_m_stride;
    kv.params[0].output_stride_h = output_key_h_stride;
    kv.params[0].count = count;
    kv.params[0].output_khm = output_key_k * output_key_h * output_key_m;
    kv.params[0].output_kh = output_key_k * output_key_h;
    kv.params[0].output_k = output_key_k;

    kv.params[1].past_ptr = past_value == nullptr ? nullptr : past_value->dptr();
    kv.params[1].ptr = value->dptr();
    kv.params[1].output_ptr = output_value->mut_dptr();
    kv.params[1].past_offset = past_value_offset;
    kv.params[1].offset = value_offset;
    kv.params[1].output_offset = output_value_offset;
    kv.params[1].past_m = past_value_m;
    kv.params[1].past_stride_b = past_value_b_stride;
    kv.params[1].past_stride_m = past_value_m_stride;
    kv.params[1].past_stride_h = past_value_h_stride;
    kv.params[1].stride_b = value_b_stride;
    kv.params[1].stride_m = value_m_stride;
    kv.params[1].stride_h = value_h_stride;
    kv.params[1].output_stride_b = output_value_b_stride;
    kv.params[1].output_stride_m = output_value_m_stride;
    kv.params[1].output_stride_h = output_value_h_stride;
    kv.params[1].count = count;
    kv.params[1].output_khm = output_value_k * output_value_h * output_value_m;
    kv.params[1].output_kh = output_value_k * output_value_h;
    kv.params[1].output_k = output_value_k;

    constexpr uint32_t block_size = 256;
    const dim3 grid_size((count - 1 + block_size) / block_size, 2);

    const int64_t elem_size = size_of_data_type * pack_size;
    cudaStream_t cuda_stream = ctx->stream()->As<ep::CudaStream>()->cuda_stream();
    if (elem_size == 16) {
      BatchConcatPastKeyValue<16, int32_t><<<grid_size, block_size, 0, cuda_stream>>>(kv);
    } else if (elem_size == 8) {
      BatchConcatPastKeyValue<8, int32_t><<<grid_size, block_size, 0, cuda_stream>>>(kv);
    } else if (elem_size == 4) {
      BatchConcatPastKeyValue<4, int32_t><<<grid_size, block_size, 0, cuda_stream>>>(kv);
    } else if (elem_size == 2) {
      BatchConcatPastKeyValue<2, int32_t><<<grid_size, block_size, 0, cuda_stream>>>(kv);
    } else if (elem_size == 1) {
      BatchConcatPastKeyValue<1, int32_t><<<grid_size, block_size, 0, cuda_stream>>>(kv);
    } else {
      UNIMPLEMENTED();
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("fused_attention_concat_past_key_value")
    .SetCreateFn<FusedAttentionConcatPastKeyValueKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA));

}  // namespace

}  // namespace user_op

}  // namespace oneflow

#endif  // WITH_CUTLASS
