
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#ifdef WITH_CUTLASS

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/cuda/cuda_stream.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/ep/include/primitive/permute.h"
#include "cutlass/arch/mma.h"
#include "cutlass/gemm/warp/mma.h"
#include "kernel_forward.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "trt_flash_attention/fmha.h"
#include "trt_flash_attention/fmha_flash_attention.h"

namespace oneflow {

namespace user_op {

namespace {

void ParseDims(const ShapeView& shape, const std::string& layout,
               const Optional<int64_t>& batch_size, const Optional<int64_t>& seq_len,
               const Optional<int64_t>& num_heads, const Optional<int64_t>& head_size,
               int64_t tensor_index, int64_t* b, int64_t* m, int64_t* h, int64_t* k,
               int64_t* b_stride, int64_t* m_stride, int64_t* h_stride, int64_t* offset,
               bool* bm_packed) {
  if (shape.NumAxes() == 2) {
    if (layout == "(BM)(HK)" || layout == "(BM)(H2K)" || layout == "(BM)(H3K)") {
      *bm_packed = true;
      CHECK(batch_size);
      CHECK(seq_len);
      *b = CHECK_JUST(batch_size);
      *m = CHECK_JUST(seq_len);
      int64_t packed_n = 0;
      if (layout == "(BM)(HK)") {
        packed_n = 1;
      } else if (layout == "(BM)(H2K)") {
        packed_n = 2;
      } else if (layout == "(BM)(H3K)") {
        packed_n = 3;
      } else {
        UNIMPLEMENTED();
      }
      const int64_t hidden_size = shape.At(1);
      if (num_heads) {
        const int64_t expected_h = CHECK_JUST(num_heads);
        const int64_t packed_h = packed_n * expected_h;
        CHECK_EQ(hidden_size % packed_h, 0);
        *h = expected_h;
        *k = hidden_size / packed_h;
      } else if (head_size) {
        const int64_t expected_k = CHECK_JUST(head_size);
        const int64_t packed_k = packed_n * expected_k;
        CHECK_EQ(hidden_size % packed_k, 0);
        *h = hidden_size / packed_k;
        *k = expected_k;
      } else {
        UNIMPLEMENTED();
      }
      *h_stride = *k * packed_n;
      *m_stride = *h_stride * *h;
      *b_stride = 0;
      if (packed_n == 1) {
        *offset = 0;
      } else if (packed_n == 2) {
        CHECK_GE(tensor_index, 1);
        *offset = (tensor_index - 1) * *k;
      } else if (packed_n == 3) {
        *offset = tensor_index * *k;
      } else {
        UNIMPLEMENTED();
      }
    } else {
      UNIMPLEMENTED();
    }
  } else if (shape.NumAxes() == 3) {
    if (layout == "BM(HK)" || layout == "BM(H2K)" || layout == "BM(H3K)" || layout == "MB(HK)"
        || layout == "MB(H2K)" || layout == "MB(H3K)") {
      *bm_packed = false;
      bool batch_first = false;
      int64_t packed_n = 0;
      const std::string layout_bm = layout.substr(0, 2);
      const std::string layout_hk = layout.substr(2);
      if (layout_bm == "BM") {
        *b = shape.At(0);
        *m = shape.At(1);
        batch_first = true;
      } else if (layout_bm == "MB") {
        *b = shape.At(1);
        *m = shape.At(0);
        batch_first = false;
      } else {
        UNIMPLEMENTED();
      }
      if (layout_hk == "(HK)") {
        packed_n = 1;
      } else if (layout_hk == "(H2K)") {
        packed_n = 2;
      } else if (layout_hk == "(H3K)") {
        packed_n = 3;
      } else {
        UNIMPLEMENTED();
      }
      const int64_t hidden_size = shape.At(2);
      if (num_heads) {
        const int64_t expected_h = CHECK_JUST(num_heads);
        const int64_t packed_h = packed_n * expected_h;
        CHECK_EQ(hidden_size % packed_h, 0);
        *h = expected_h;
        *k = hidden_size / packed_h;
      } else if (head_size) {
        const int64_t expected_k = CHECK_JUST(head_size);
        const int64_t packed_k = packed_n * expected_k;
        CHECK_EQ(hidden_size % packed_k, 0);
        *h = hidden_size / packed_k;
        *k = expected_k;
      } else {
        UNIMPLEMENTED();
      }
      *h_stride = *k * packed_n;
      if (batch_first) {
        *m_stride = *h_stride * *h;
        *b_stride = *m_stride * *m;
      } else {
        *b_stride = *h_stride * *h;
        *m_stride = *b_stride * *b;
      }
      if (packed_n == 1) {
        *offset = 0;
      } else if (packed_n == 2) {
        CHECK_GE(tensor_index, 1);
        *offset = (tensor_index - 1) * *k;
      } else if (packed_n == 3) {
        *offset = tensor_index * *k;
      } else {
        UNIMPLEMENTED();
      }
    } else if (layout == "(BM)HK") {
      *bm_packed = true;
      CHECK(batch_size);
      CHECK(seq_len);
      *b = CHECK_JUST(batch_size);
      *m = CHECK_JUST(seq_len);
      *h = shape.At(1);
      *k = shape.At(2);
      *h_stride = *k;
      *m_stride = *h_stride * *h;
      *b_stride = 0;
    } else {
      UNIMPLEMENTED();
    }
  } else if (shape.NumAxes() == 4) {
    *bm_packed = false;
    if (layout == "BMHK") {
      *b = shape.At(0);
      *m = shape.At(1);
      *h = shape.At(2);
      *k = shape.At(3);
      *h_stride = *k;
      *m_stride = *h_stride * *h;
      *b_stride = *m_stride * *m;
    } else if (layout == "BHMK") {
      *b = shape.At(0);
      *m = shape.At(2);
      *h = shape.At(1);
      *k = shape.At(3);
      *m_stride = *k;
      *h_stride = *m_stride * *m;
      *b_stride = *h_stride * *h;
    } else if (layout == "MBHK") {
      *b = shape.At(1);
      *m = shape.At(0);
      *h = shape.At(2);
      *k = shape.At(3);
      *h_stride = *k;
      *b_stride = *h_stride * *h;
      *m_stride = *b_stride * *b;
    } else {
      UNIMPLEMENTED();
    }
    *offset = 0;
  } else {
    UNIMPLEMENTED();
  };
  if (batch_size) {
    const int64_t expected_b = CHECK_JUST(batch_size);
    CHECK_EQ(*b, expected_b);
  }
  if (seq_len) {
    const int64_t expected_m = CHECK_JUST(seq_len);
    CHECK_EQ(*m, expected_m);
  }
  if (num_heads) {
    const int64_t expected_h = CHECK_JUST(num_heads);
    CHECK_EQ(*h, expected_h);
  }
  if (head_size) {
    const int64_t expected_k = CHECK_JUST(head_size);
    CHECK_EQ(*k, expected_k);
  }
}

void ParseDims(const ShapeView& shape, const std::string& layout,
               const Optional<int64_t>& num_heads, const Optional<int64_t>& head_size,
               int64_t tensor_index, int64_t* b, int64_t* m, int64_t* h, int64_t* k,
               int64_t* b_stride, int64_t* m_stride, int64_t* h_stride, int64_t* offset) {
  bool bm_packed{};
  ParseDims(shape, layout, Optional<int64_t>(), Optional<int64_t>(), num_heads, head_size,
            tensor_index, b, m, h, k, b_stride, m_stride, h_stride, offset, &bm_packed);
}

template<typename T, int pack_size>
struct alignas(pack_size * sizeof(T)) Pack {
  T elem[pack_size];
};

template<typename T>
__global__ void PackQkv(int b, int s, int nh, int d, const T* q, const T* k, const T* v, T* o,
                        int32_t* seq_len) {
  int count = b * s * nh * d * 3;
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < count; i += blockDim.x * gridDim.x) {
    int row = i / (d * 3);
    int out_col = i - row * (d * 3);
    T out;
    if (out_col < d) {
      out = q[row * d + out_col];
    } else if (out_col < 2 * d) {
      out = k[row * d + out_col - d];
    } else {
      out = v[row * d + out_col - d * 2];
    }
    o[i] = out;
  }
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < b + 1; i += blockDim.x * gridDim.x) {
    seq_len[i] = i * s;
  }
}

struct Params {
  DataType data_type;
  int64_t num_batches;
  int64_t num_heads;
  int64_t query_seq_len;
  int64_t kv_seq_len;
  int64_t head_size;
  int64_t value_head_size;
  int64_t q_stride_b;
  int64_t q_stride_m;
  int64_t q_stride_h;
  int64_t k_stride_b;
  int64_t k_stride_m;
  int64_t k_stride_h;
  int64_t v_stride_b;
  int64_t v_stride_m;
  int64_t v_stride_h;
  std::string attn_mask_type;
  int64_t causal_diagonal_offset;
  const void* query_ptr;
  const void* key_ptr;
  const void* value_ptr;
  const void* attn_bias_ptr;
  const void* query_seq_start_ptr;
  const void* key_seq_start_ptr;
  const void* key_seq_len_ptr;
  int64_t attn_bias_stride_b;
  int64_t attn_bias_stride_h;
  int64_t attn_bias_stride_m;
  void* out_ptr;
  void* workspace;
  int64_t workspace_size;
  float scale;
};

template<typename T, typename ArchTag, bool is_aligned, int queries_per_block, int keys_per_block,
         bool single_value_iteration, bool with_attn_bias>
void LaunchCutlassFmha(const Params& params, ep::CudaStream* stream) {
  // The fmha implementation below is based on xformers's fmha
  // implementation at:
  // https://github.com/facebookresearch/xformers/tree/main/xformers/csrc/attention/cuda/fmha
  using Attention = AttentionKernel<T, ArchTag, is_aligned, queries_per_block, keys_per_block,
                                    single_value_iteration, false, with_attn_bias>;
  typename Attention::Params p{};
  p.query_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.query_ptr));
  p.key_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.key_ptr));
  p.value_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.value_ptr));
  p.attn_bias_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.attn_bias_ptr));
  p.seqstart_q_ptr =
      const_cast<int32_t*>(reinterpret_cast<const int32_t*>(params.query_seq_start_ptr));
  p.seqstart_k_ptr =
      const_cast<int32_t*>(reinterpret_cast<const int32_t*>(params.key_seq_start_ptr));
  p.seqlen_k_ptr = const_cast<int32_t*>(reinterpret_cast<const int32_t*>(params.key_seq_len_ptr));
  p.logsumexp_ptr = nullptr;
  p.output_ptr = reinterpret_cast<T*>(params.out_ptr);
  if (Attention::kNeedsOutputAccumulatorBuffer) {
    using Acc = typename Attention::accum_t;
    CHECK_GE(params.workspace_size, params.num_batches * params.query_seq_len * params.num_heads
                                        * params.value_head_size * sizeof(Acc));
    p.output_accum_ptr = reinterpret_cast<Acc*>(params.workspace);
  } else {
    p.output_accum_ptr = nullptr;
  }
  p.num_heads = params.num_heads;
  p.num_batches = params.num_batches;
  p.head_dim = params.head_size;
  p.head_dim_value = params.value_head_size;
  p.num_queries = params.query_seq_len;
  p.num_keys = params.kv_seq_len;
  p.q_strideM = params.q_stride_m;
  p.k_strideM = params.k_stride_m;
  p.v_strideM = params.v_stride_m;
  p.o_strideM = p.head_dim_value * p.num_heads;
  p.bias_strideM = params.attn_bias_stride_m;

  p.q_strideH = params.q_stride_h;
  p.k_strideH = params.k_stride_h;
  p.v_strideH = params.v_stride_h;
  p.bias_strideH = params.attn_bias_stride_h;

  p.q_strideB = params.q_stride_b;
  p.k_strideB = params.k_stride_b;
  p.v_strideB = params.v_stride_b;
  p.bias_strideB = params.attn_bias_stride_b;

  p.scale = params.scale;

  if (params.attn_mask_type == "none") {
    p.custom_mask_type = Attention::NoCustomMask;
  } else if (params.attn_mask_type == "causal_from_top_left") {
    p.custom_mask_type = Attention::CausalFromTopLeft;
  } else if (params.attn_mask_type == "causal_from_bottom_right") {
    p.custom_mask_type = Attention::CausalFromBottomRight;
  } else {
    UNIMPLEMENTED();
  }
  p.causal_diagonal_offset = params.causal_diagonal_offset;
  p.use_dropout = false;

  constexpr auto kernel_fn = attention_kernel_batched_impl<Attention>;
  int smem_bytes = sizeof(typename Attention::SharedStorage);
  if (smem_bytes > 0xc000) {
    static bool once = [&]() {
      cudaFuncSetAttribute(kernel_fn, cudaFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
      return true;
    }();
  }
  CHECK(Attention::check_supported(p));
  kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes, stream->cuda_stream()>>>(p);
}

template<typename T, typename ArchTag, bool is_aligned, int queries_per_block, int keys_per_block,
         bool single_value_iteration>
void DispatchWithAttnBias(const Params& params, ep::CudaStream* stream) {
  if (params.attn_bias_ptr != nullptr) {
    LaunchCutlassFmha<T, ArchTag, is_aligned, queries_per_block, keys_per_block,
                      single_value_iteration, true>(params, stream);
  } else {
    LaunchCutlassFmha<T, ArchTag, is_aligned, queries_per_block, keys_per_block,
                      single_value_iteration, false>(params, stream);
  }
}

template<typename T, typename ArchTag, bool is_aligned, int queries_per_block, int keys_per_block>
void DispatchSingleValueIteration(const Params& params, ep::CudaStream* stream) {
  if (params.value_head_size <= keys_per_block) {
    DispatchWithAttnBias<T, ArchTag, is_aligned, queries_per_block, keys_per_block, true>(params,
                                                                                          stream);
  } else {
    DispatchWithAttnBias<T, ArchTag, is_aligned, queries_per_block, keys_per_block, false>(params,
                                                                                           stream);
  }
}

template<typename T, typename ArchTag, bool is_aligned>
void DispatchKeysPerBlock(const Params& params, ep::CudaStream* stream) {
  if (params.value_head_size <= 64) {
    DispatchSingleValueIteration<T, ArchTag, is_aligned, 64, 64>(params, stream);
  } else {
    DispatchSingleValueIteration<T, ArchTag, is_aligned, 32, 128>(params, stream);
  }
}

template<typename T, typename ArchTag>
void DispatchIsAligned(const Params& params, ep::CudaStream* stream) {
  if (reinterpret_cast<uintptr_t>(params.query_ptr) % 16 == 0
      && reinterpret_cast<uintptr_t>(params.key_ptr) % 16 == 0
      && reinterpret_cast<uintptr_t>(params.value_ptr) % 16 == 0
      && params.attn_bias_stride_m % (16 / sizeof(T)) == 0
      && params.head_size % (16 / sizeof(T)) == 0
      && params.value_head_size % (16 / sizeof(T)) == 0) {
    DispatchKeysPerBlock<T, ArchTag, true>(params, stream);
  } else {
    DispatchKeysPerBlock<T, ArchTag, false>(params, stream);
  }
}

template<typename T>
void DispatchArchTag(const Params& params, ep::CudaStream* stream) {
  const int major = stream->device_properties().major;
  const int minor = stream->device_properties().minor;

  if (major == 8) {
    DispatchIsAligned<T, cutlass::arch::Sm80>(params, stream);
  } else if (major == 7) {
    if (minor == 5) {
      DispatchIsAligned<T, cutlass::arch::Sm75>(params, stream);
    } else {
      DispatchIsAligned<T, cutlass::arch::Sm70>(params, stream);
    }
  } else {
    UNIMPLEMENTED();
  }
}

void DispatchCutlassFmha(const Params& params, ep::CudaStream* stream) {
  if (params.data_type == DataType::kFloat16) {
    DispatchArchTag<cutlass::half_t>(params, stream);
  } else if (params.data_type == DataType::kFloat) {
    DispatchArchTag<float>(params, stream);
  } else {
    UNIMPLEMENTED();
  }
}

class FusedMultiHeadAttentionInferenceKernel final : public user_op::OpKernel,
                                                     public user_op::CudaGraphSupport {
 public:
  FusedMultiHeadAttentionInferenceKernel() = default;
  ~FusedMultiHeadAttentionInferenceKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const Tensor* query = ctx->Tensor4ArgNameAndIndex("query", 0);
    const Tensor* key = ctx->Tensor4ArgNameAndIndex("key", 0);
    const Tensor* value = ctx->Tensor4ArgNameAndIndex("value", 0);
    const Tensor* attn_bias = nullptr;
    if (ctx->has_input("attn_bias", 0)) { attn_bias = ctx->Tensor4ArgNameAndIndex("attn_bias", 0); }
    const Tensor* query_seq_start = nullptr;
    const Tensor* key_seq_start = nullptr;
    const Tensor* key_seq_len = nullptr;
    const float scale = ctx->Attr<double>("scale");
    if (ctx->has_input("query_seq_start", 0)) {
      CHECK(ctx->has_input("key_seq_start", 0));
      query_seq_start = ctx->Tensor4ArgNameAndIndex("query_seq_start", 0);
      key_seq_start = ctx->Tensor4ArgNameAndIndex("key_seq_start", 0);
      CHECK(query_seq_start->data_type() == DataType::kInt32);
      CHECK(key_seq_start->data_type() == DataType::kInt32);
      CHECK_EQ(query_seq_start->shape_view().NumAxes(), 1);
      CHECK_GT(query_seq_start->shape_view().At(0), 1);
      CHECK(query_seq_start->shape_view() == key_seq_start->shape_view());
      if (ctx->has_input("key_seq_len", 0)) {
        key_seq_len = ctx->Tensor4ArgNameAndIndex("key_seq_len", 0);
        CHECK(key_seq_len->data_type() == DataType::kInt32);
        CHECK_EQ(key_seq_len->shape_view().NumAxes(), 1);
        CHECK_EQ(key_seq_len->shape_view().At(0), query_seq_start->shape_view().At(0) - 1);
      }
    } else {
      CHECK(!ctx->has_input("key_seq_start", 0));
      CHECK(!ctx->has_input("key_seq_len", 0));
    }
    Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    Tensor* tmp = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const DataType data_type = query->data_type();
    CHECK_EQ(key->data_type(), data_type);
    CHECK_EQ(value->data_type(), data_type);
    CHECK_EQ(out->data_type(), data_type);
    const int64_t query_head_size = ctx->Attr<int64_t>("query_head_size");
    const std::string& attn_mask_type = ctx->Attr<std::string>("attn_mask_type");
    const int64_t causal_diagonal_offset = ctx->Attr<int64_t>("causal_diagonal_offset");
    CHECK_GE(causal_diagonal_offset, 0);
    const std::string& query_layout = ctx->Attr<std::string>("query_layout");
    const std::string& key_layout = ctx->Attr<std::string>("key_layout");
    const std::string& value_layout = ctx->Attr<std::string>("value_layout");
    const std::string& output_layout = ctx->Attr<std::string>("output_layout");

    Optional<int64_t> batch_size;
    if (query_seq_start != nullptr) { batch_size = query_seq_start->shape_view().At(0) - 1; }
    Optional<int64_t> query_max_seq_len;
    const int64_t attr_query_max_seq_len = ctx->Attr<int64_t>("query_max_seq_len");
    if (attr_query_max_seq_len != 0) { query_max_seq_len = attr_query_max_seq_len; }
    Optional<int64_t> key_max_seq_len;
    const int64_t attr_key_max_seq_len = ctx->Attr<int64_t>("key_max_seq_len");
    if (attr_key_max_seq_len != 0) { key_max_seq_len = attr_key_max_seq_len; }

    int64_t q_b = 0;
    int64_t q_m = 0;
    int64_t q_h = 0;
    int64_t q_k = 0;
    int64_t q_b_stride = 0;
    int64_t q_m_stride = 0;
    int64_t q_h_stride = 0;
    int64_t q_offset = 0;
    bool q_bm_packed = false;
    ParseDims(query->shape_view(), query_layout, batch_size, query_max_seq_len, Optional<int64_t>(),
              query_head_size, 0, &q_b, &q_m, &q_h, &q_k, &q_b_stride, &q_m_stride, &q_h_stride,
              &q_offset, &q_bm_packed);
    if (q_bm_packed) { CHECK(query_seq_start != nullptr); }

    int64_t k_b = 0;
    int64_t k_m = 0;
    int64_t k_h = 0;
    int64_t k_k = 0;
    int64_t k_b_stride = 0;
    int64_t k_m_stride = 0;
    int64_t k_h_stride = 0;
    int64_t k_offset = 0;
    bool k_bm_packed = false;
    ParseDims(key->shape_view(), key_layout, q_b, key_max_seq_len, Optional<int64_t>(),
              query_head_size, 1, &k_b, &k_m, &k_h, &k_k, &k_b_stride, &k_m_stride, &k_h_stride,
              &k_offset, &k_bm_packed);
    CHECK_EQ(k_b, q_b);
    CHECK_EQ(k_h, q_h);
    CHECK_EQ(k_bm_packed, q_bm_packed);

    int64_t v_b = 0;
    int64_t v_m = 0;
    int64_t v_h = 0;
    int64_t v_k = 0;
    int64_t v_b_stride = 0;
    int64_t v_m_stride = 0;
    int64_t v_h_stride = 0;
    int64_t v_offset = 0;
    bool v_bm_packed = false;
    ParseDims(value->shape_view(), value_layout, q_b, k_m, q_h, Optional<int64_t>(), 2, &v_b, &v_m,
              &v_h, &v_k, &v_b_stride, &v_m_stride, &v_h_stride, &v_offset, &v_bm_packed);
    CHECK_EQ(v_b, q_b);
    CHECK_EQ(v_m, k_m);
    CHECK_EQ(v_bm_packed, k_bm_packed);
    if (output_layout == "BM(HK)") {
      CHECK(!q_bm_packed);
      CHECK_EQ(out->shape_view().NumAxes(), 3);
      CHECK_EQ(out->shape_view().At(0), q_b);
      CHECK_EQ(out->shape_view().At(1), q_m);
      CHECK_EQ(out->shape_view().At(2), q_h * v_k);
    } else if (output_layout == "MB(HK)") {
      CHECK(!q_bm_packed);
      CHECK_EQ(out->shape_view().NumAxes(), 3);
      CHECK_EQ(q_b, 1);
      CHECK_EQ(out->shape_view().At(0), q_m);
      CHECK_EQ(out->shape_view().At(1), q_b);
      CHECK_EQ(out->shape_view().At(2), q_h * v_k);
    } else if (output_layout == "(BM)(HK)") {
      CHECK(q_bm_packed);
      CHECK_EQ(out->shape_view().NumAxes(), 2);
      CHECK_EQ(out->shape_view().At(0), query->shape_view().At(0));
      CHECK_EQ(out->shape_view().At(1), q_h * v_k);
    } else {
      UNIMPLEMENTED();
    }

    auto* cuda_stream = ctx->stream()->As<ep::CudaStream>();

    // Compatible with typo `KERENL`
    const bool enable_trt_flash_attn =
        ParseBooleanFromEnv(
            "ONEFLOW_KERNEL_FMHA_ENABLE_TRT_FLASH_ATTN_IMPL",
            ParseBooleanFromEnv("ONEFLOW_KERENL_FMHA_ENABLE_TRT_FLASH_ATTN_IMPL", true))
        && ParseBooleanFromEnv("ONEFLOW_MATMUL_ALLOW_HALF_PRECISION_ACCUMULATION", false);
    const bool is_default_scale =
        std::abs(scale - 1.0 / std::sqrt(static_cast<float>(q_k))) <= 1e-5;
    const int arch = cuda_stream->cuda_arch() / 10;
    const bool is_trt_supported_arch = (arch == 75 || arch == 80 || arch == 86 || arch == 89);
    const bool is_trt_supported_head_size = ((q_k == 40) || (q_k == 64));
    // Avoid PackQKV overhead when seq_len is small.
    const bool is_long_seq_len = q_m >= 512;
    const bool is_trt_supported_layout = (query_layout == "BMHK" || query_layout == "BM(HK)")
                                         && (key_layout == "BMHK" || key_layout == "BM(HK)")
                                         && (value_layout == "BMHK" || value_layout == "BM(HK)")
                                         && (output_layout == "BMHK" || output_layout == "BM(HK)");
    if (is_default_scale && query_seq_start == nullptr && enable_trt_flash_attn
        && data_type == DataType::kFloat16 && q_m == k_m && q_k == v_k && is_trt_supported_head_size
        && is_long_seq_len && is_trt_supported_arch && attn_mask_type == "none"
        && attn_bias == nullptr && is_trt_supported_layout) {
      // The fmha implementation below is based on TensorRT's multiHeadFlashAttentionPlugin
      // implementation at:
      // https://github.com/NVIDIA/TensorRT/tree/main/plugin/multiHeadFlashAttentionPlugin
      int32_t cu_seqlens_d_size = (q_b + 1) * sizeof(int32_t);
      int32_t* cu_seqlens_d = reinterpret_cast<int32_t*>(tmp->mut_dptr());
      half* packed_qkv =
          reinterpret_cast<half*>(tmp->mut_dptr<char>() + GetCudaAlignedSize(cu_seqlens_d_size));
      constexpr int pack_size = 4;
      using PackType = Pack<half, pack_size>;
      const int64_t count = q_b * q_m * q_h * q_k * 3 / pack_size;
      PackQkv<PackType><<<(count - 1 + 256) / 256, 256, 0, cuda_stream->cuda_stream()>>>(
          q_b, q_m, q_h, q_k / pack_size, reinterpret_cast<const PackType*>(query->dptr()),
          reinterpret_cast<const PackType*>(key->dptr()),
          reinterpret_cast<const PackType*>(value->dptr()), reinterpret_cast<PackType*>(packed_qkv),
          cu_seqlens_d);

#ifdef WITH_CUDA_GRAPHS
      cudaStreamCaptureMode mode = cudaStreamCaptureModeRelaxed;
      if (cuda_stream->IsGraphCapturing()) {
        OF_CUDA_CHECK(cudaThreadExchangeStreamCaptureMode(&mode));
      }
#endif  // WITH_CUDA_GRAPHS
      nvinfer1::plugin::FusedMultiHeadFlashAttentionKernel const* kernels =
          nvinfer1::plugin::getFMHAFlashCubinKernels(nvinfer1::plugin::DATA_TYPE_FP16, arch);
#ifdef WITH_CUDA_GRAPHS
      if (cuda_stream->IsGraphCapturing()) {
        OF_CUDA_CHECK(cudaThreadExchangeStreamCaptureMode(&mode));
      }
#endif  // WITH_CUDA_GRAPHS
      nvinfer1::plugin::runFMHFAKernel(packed_qkv, cu_seqlens_d, out->mut_dptr(), q_b * q_m, arch,
                                       kernels, q_b, q_h, q_k, q_m, cuda_stream->cuda_stream());
      return;
    }

    Params params{};
    params.data_type = data_type;
    params.num_batches = q_b;
    params.num_heads = q_h;
    params.query_seq_len = q_m;
    params.kv_seq_len = k_m;
    params.head_size = q_k;
    params.value_head_size = v_k;
    params.scale = scale;
    params.q_stride_b = q_b_stride;
    params.q_stride_m = q_m_stride;
    params.q_stride_h = q_h_stride;
    params.k_stride_b = k_b_stride;
    params.k_stride_m = k_m_stride;
    params.k_stride_h = k_h_stride;
    params.v_stride_b = v_b_stride;
    params.v_stride_m = v_m_stride;
    params.v_stride_h = v_h_stride;
    params.query_ptr = query->dptr<char>() + q_offset * GetSizeOfDataType(data_type);
    params.key_ptr = key->dptr<char>() + k_offset * GetSizeOfDataType(data_type);
    params.value_ptr = value->dptr<char>() + v_offset * GetSizeOfDataType(data_type);
    params.query_seq_start_ptr =
        query_seq_start == nullptr ? nullptr : query_seq_start->dptr<int32_t>();
    params.key_seq_start_ptr = key_seq_start == nullptr ? nullptr : key_seq_start->dptr<int32_t>();
    params.key_seq_len_ptr = key_seq_len == nullptr ? nullptr : key_seq_len->dptr<int32_t>();
    params.out_ptr = out->mut_dptr();
    const int64_t tmp_buffer_size = tmp->shape_view().elem_cnt();
    params.workspace = tmp->mut_dptr();
    params.workspace_size = tmp_buffer_size;
    params.attn_mask_type = attn_mask_type;
    params.causal_diagonal_offset = causal_diagonal_offset;
    if (attn_bias != nullptr) {
      const int64_t num_attn_bias_axes = attn_bias->shape_view().NumAxes();
      CHECK_GE(num_attn_bias_axes, 1);
      CHECK_LE(num_attn_bias_axes, 4);
      DimVector padded_attn_bias_shape;
      for (int i = 0; i < 4 - num_attn_bias_axes; ++i) { padded_attn_bias_shape.push_back(1); }
      for (int i = 0; i < num_attn_bias_axes; ++i) {
        padded_attn_bias_shape.push_back(attn_bias->shape_view().At(i));
      }
      CHECK_GE(padded_attn_bias_shape.at(3), k_m);
      int64_t bias_stride = padded_attn_bias_shape.at(3);
      if (padded_attn_bias_shape.at(2) == 1) {
        params.attn_bias_stride_m = 0;
      } else {
        CHECK_GE(padded_attn_bias_shape.at(2), q_m);
        params.attn_bias_stride_m = bias_stride;
        bias_stride *= padded_attn_bias_shape.at(2);
      }
      if (padded_attn_bias_shape.at(1) == 1) {
        params.attn_bias_stride_h = 0;
      } else {
        CHECK_EQ(padded_attn_bias_shape.at(1), q_h);
        params.attn_bias_stride_h = bias_stride;
        bias_stride *= q_h;
      }
      if (padded_attn_bias_shape.at(0) == 1) {
        params.attn_bias_stride_b = 0;
      } else {
        CHECK_EQ(padded_attn_bias_shape.at(0), q_b);
        params.attn_bias_stride_b = bias_stride;
      }
      params.attn_bias_ptr = attn_bias->dptr();
    } else {
      params.attn_bias_ptr = nullptr;
      params.attn_bias_stride_m = 0;
      params.attn_bias_stride_h = 0;
      params.attn_bias_stride_b = 0;
    }
    DispatchCutlassFmha(params, cuda_stream);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

size_t InferTmpBufferSize(InferContext* ctx) {
  const auto& out_desc = ctx->OutputTensorDesc("out", 0);
  size_t buffer_size = 0;
  buffer_size +=
      GetCudaAlignedSize(out_desc.shape().elem_cnt() * GetSizeOfDataType(DataType::kFloat));
  buffer_size +=
      GetCudaAlignedSize(out_desc.shape().elem_cnt() * GetSizeOfDataType(out_desc.data_type())) * 3;
  buffer_size +=
      GetCudaAlignedSize((out_desc.shape().At(0) + 1) * GetSizeOfDataType(DataType::kInt32));
  return buffer_size;
}

#define REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(dtype)    \
  REGISTER_USER_KERNEL("fused_multi_head_attention_inference")         \
      .SetCreateFn<FusedMultiHeadAttentionInferenceKernel>()           \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("out", 0) == dtype))   \
      .SetInferTmpSizeFn(InferTmpBufferSize);

REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(DataType::kFloat16)
REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(DataType::kFloat)

template<typename Index>
struct ConcatParam {
  const void* past_ptr;
  const void* ptr;
  void* output_ptr;
  Index past_offset;
  Index offset;
  Index output_offset;
  Index past_m;
  Index past_stride_b;
  Index past_stride_m;
  Index past_stride_h;
  Index stride_b;
  Index stride_m;
  Index stride_h;
  Index output_stride_b;
  Index output_stride_m;
  Index output_stride_h;
  Index count;
  Index output_khm;
  Index output_kh;
  Index output_k;
};

template<typename Index>
struct BatchConcatParam {
  ConcatParam<Index> params[2];
};

template<typename T, typename Index>
__device__ void ConcatPastKeyValue(ConcatParam<Index> p) {
  for (Index i = blockIdx.x * blockDim.x + threadIdx.x; i < p.count; i += blockDim.x * gridDim.x) {
    Index b_idx = i / p.output_khm;
    Index b_off = i - b_idx * p.output_khm;
    Index m_idx = b_off / p.output_kh;
    Index m_off = b_off - m_idx * p.output_kh;
    Index h_idx = m_off / p.output_k;
    Index k_idx = m_off - h_idx * p.output_k;
    T v;
    if (m_idx < p.past_m) {
      v = reinterpret_cast<const T*>(
          p.past_ptr)[p.past_offset + b_idx * p.past_stride_b + m_idx * p.past_stride_m
                      + h_idx * p.past_stride_h + k_idx];
    } else {
      v = reinterpret_cast<const T*>(
          p.ptr)[p.offset + b_idx * p.stride_b + (m_idx - p.past_m) * p.stride_m
                 + h_idx * p.stride_h + k_idx];
    }
    reinterpret_cast<T*>(
        p.output_ptr)[p.output_offset + b_idx * p.output_stride_b + m_idx * p.output_stride_m
                      + h_idx * p.output_stride_h + k_idx] = v;
  }
}

template<size_t elem_size, typename Index>
__global__ void BatchConcatPastKeyValue(BatchConcatParam<Index> params) {
  if (blockIdx.y == 0) {
    ConcatPastKeyValue<std::aligned_storage<elem_size, elem_size>::type, Index>(params.params[0]);
  } else if (blockIdx.y == 1) {
    ConcatPastKeyValue<std::aligned_storage<elem_size, elem_size>::type, Index>(params.params[1]);
  } else {
    // do nothing
  }
}

class FusedAttentionConcatPastKeyValueKernel final : public user_op::OpKernel,
                                                     public user_op::CudaGraphSupport {
 public:
  FusedAttentionConcatPastKeyValueKernel() = default;
  ~FusedAttentionConcatPastKeyValueKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const Tensor* key = ctx->Tensor4ArgNameAndIndex("key", 0);
    const Tensor* value = ctx->Tensor4ArgNameAndIndex("value", 0);
    Tensor* output_key = ctx->Tensor4ArgNameAndIndex("output_key", 0);
    Tensor* output_value = ctx->Tensor4ArgNameAndIndex("output_value", 0);
    const DataType data_type = key->data_type();
    const Tensor* past_key = nullptr;
    const Tensor* past_value = nullptr;
    if (ctx->has_input("past_key", 0)) {
      CHECK(ctx->has_input("past_value", 0));
      past_key = ctx->Tensor4ArgNameAndIndex("past_key", 0);
      past_value = ctx->Tensor4ArgNameAndIndex("past_value", 0);
      CHECK_EQ(past_key->data_type(), data_type);
      CHECK_EQ(past_value->data_type(), data_type);
    } else {
      CHECK(!ctx->has_input("past_value", 0));
    }
    CHECK_EQ(value->data_type(), data_type);
    CHECK_EQ(output_key->data_type(), data_type);
    CHECK_EQ(output_value->data_type(), data_type);
    const int64_t size_of_data_type = GetSizeOfDataType(data_type);
    const int64_t key_head_size = ctx->Attr<int64_t>("key_head_size");
    const std::string& past_key_layout = ctx->Attr<std::string>("past_key_layout");
    const std::string& past_value_layout = ctx->Attr<std::string>("past_value_layout");
    const std::string& key_layout = ctx->Attr<std::string>("key_layout");
    const std::string& value_layout = ctx->Attr<std::string>("value_layout");

    int64_t pack_size = 16 / size_of_data_type;
    while (key_head_size % pack_size != 0) { pack_size /= 2; }

    auto ParsePackedDims =
        [](const ShapeView& shape, const std::string& layout, const Optional<int64_t>& num_heads,
           const Optional<int64_t>& head_size, int64_t tensor_index, int64_t* b, int64_t* m,
           int64_t* h, int64_t* k, int64_t* b_stride, int64_t* m_stride, int64_t* h_stride,
           int64_t* offset, int64_t pack_size) {
          ParseDims(shape, layout, num_heads, head_size, tensor_index, b, m, h, k, b_stride,
                    m_stride, h_stride, offset);
          *k /= pack_size;
          *b_stride /= pack_size;
          *m_stride /= pack_size;
          *h_stride /= pack_size;
          *offset /= pack_size;
        };

    int64_t key_b = 0;
    int64_t key_m = 0;
    int64_t key_h = 0;
    int64_t key_k = 0;
    int64_t key_b_stride = 0;
    int64_t key_m_stride = 0;
    int64_t key_h_stride = 0;
    int64_t key_offset = 0;
    ParsePackedDims(key->shape_view(), key_layout, Optional<int64_t>(), key_head_size, 1, &key_b,
                    &key_m, &key_h, &key_k, &key_b_stride, &key_m_stride, &key_h_stride,
                    &key_offset, pack_size);

    int64_t value_b = 0;
    int64_t value_m = 0;
    int64_t value_h = 0;
    int64_t value_k = 0;
    int64_t value_b_stride = 0;
    int64_t value_m_stride = 0;
    int64_t value_h_stride = 0;
    int64_t value_offset = 0;
    ParsePackedDims(value->shape_view(), value_layout, key_h, key_head_size, 2, &value_b, &value_m,
                    &value_h, &value_k, &value_b_stride, &value_m_stride, &value_h_stride,
                    &value_offset, pack_size);
    CHECK_EQ(value_b, key_b);
    CHECK_EQ(value_m, key_m);

    int64_t past_key_b = 0;
    int64_t past_key_m = 0;
    int64_t past_key_h = 0;
    int64_t past_key_k = 0;
    int64_t past_key_b_stride = 0;
    int64_t past_key_m_stride = 0;
    int64_t past_key_h_stride = 0;
    int64_t past_key_offset = 0;
    if (past_key != nullptr) {
      ParsePackedDims(past_key->shape_view(), past_key_layout, key_h, key_head_size, 1, &past_key_b,
                      &past_key_m, &past_key_h, &past_key_k, &past_key_b_stride, &past_key_m_stride,
                      &past_key_h_stride, &past_key_offset, pack_size);
    }

    int64_t past_value_b = 0;
    int64_t past_value_m = 0;
    int64_t past_value_h = 0;
    int64_t past_value_k = 0;
    int64_t past_value_b_stride = 0;
    int64_t past_value_m_stride = 0;
    int64_t past_value_h_stride = 0;
    int64_t past_value_offset = 0;
    if (past_value != nullptr) {
      ParsePackedDims(past_value->shape_view(), past_value_layout, key_h, key_head_size, 2,
                      &past_value_b, &past_value_m, &past_value_h, &past_value_k,
                      &past_value_b_stride, &past_value_m_stride, &past_value_h_stride,
                      &past_value_offset, pack_size);
    }
    CHECK_EQ(past_value_b, past_key_b);
    CHECK_EQ(past_value_m, past_key_m);

    int64_t output_key_b = 0;
    int64_t output_key_m = 0;
    int64_t output_key_h = 0;
    int64_t output_key_k = 0;
    int64_t output_key_b_stride = 0;
    int64_t output_key_m_stride = 0;
    int64_t output_key_h_stride = 0;
    int64_t output_key_offset = 0;
    ParsePackedDims(output_key->shape_view(), past_key_layout, key_h, key_head_size, 1,
                    &output_key_b, &output_key_m, &output_key_h, &output_key_k,
                    &output_key_b_stride, &output_key_m_stride, &output_key_h_stride,
                    &output_key_offset, pack_size);
    CHECK_EQ(output_key_b, key_b);
    CHECK_EQ(output_key_m, past_key_m + key_m);

    int64_t output_value_b = 0;
    int64_t output_value_m = 0;
    int64_t output_value_h = 0;
    int64_t output_value_k = 0;
    int64_t output_value_b_stride = 0;
    int64_t output_value_m_stride = 0;
    int64_t output_value_h_stride = 0;
    int64_t output_value_offset = 0;
    ParsePackedDims(output_value->shape_view(), past_value_layout, key_h, key_head_size, 2,
                    &output_value_b, &output_value_m, &output_value_h, &output_value_k,
                    &output_value_b_stride, &output_value_m_stride, &output_value_h_stride,
                    &output_value_offset, pack_size);
    CHECK_EQ(output_value_b, key_b);
    CHECK_EQ(output_value_m, past_value_m + value_m);

    int64_t max_tensor_elem = (1 << 30) * pack_size;
    CHECK((past_key == nullptr || past_key->shape_view().elem_cnt() <= max_tensor_elem)
          && (past_value == nullptr || past_value->shape_view().elem_cnt() <= max_tensor_elem)
          && key->shape_view().elem_cnt() <= max_tensor_elem
          && value->shape_view().elem_cnt() <= max_tensor_elem
          && output_key->shape_view().elem_cnt() <= max_tensor_elem
          && output_value->shape_view().elem_cnt() <= max_tensor_elem);

    int64_t count = output_key_b * output_key_m * output_key_h * output_key_k;
    BatchConcatParam<int32_t> kv;

    kv.params[0].past_ptr = past_key == nullptr ? nullptr : past_key->dptr();
    kv.params[0].ptr = key->dptr();
    kv.params[0].output_ptr = output_key->mut_dptr();
    kv.params[0].past_offset = past_key_offset;
    kv.params[0].offset = key_offset;
    kv.params[0].output_offset = output_key_offset;
    kv.params[0].past_m = past_key_m;
    kv.params[0].past_stride_b = past_key_b_stride;
    kv.params[0].past_stride_m = past_key_m_stride;
    kv.params[0].past_stride_h = past_key_h_stride;
    kv.params[0].stride_b = key_b_stride;
    kv.params[0].stride_m = key_m_stride;
    kv.params[0].stride_h = key_h_stride;
    kv.params[0].output_stride_b = output_key_b_stride;
    kv.params[0].output_stride_m = output_key_m_stride;
    kv.params[0].output_stride_h = output_key_h_stride;
    kv.params[0].count = count;
    kv.params[0].output_khm = output_key_k * output_key_h * output_key_m;
    kv.params[0].output_kh = output_key_k * output_key_h;
    kv.params[0].output_k = output_key_k;

    kv.params[1].past_ptr = past_value == nullptr ? nullptr : past_value->dptr();
    kv.params[1].ptr = value->dptr();
    kv.params[1].output_ptr = output_value->mut_dptr();
    kv.params[1].past_offset = past_value_offset;
    kv.params[1].offset = value_offset;
    kv.params[1].output_offset = output_value_offset;
    kv.params[1].past_m = past_value_m;
    kv.params[1].past_stride_b = past_value_b_stride;
    kv.params[1].past_stride_m = past_value_m_stride;
    kv.params[1].past_stride_h = past_value_h_stride;
    kv.params[1].stride_b = value_b_stride;
    kv.params[1].stride_m = value_m_stride;
    kv.params[1].stride_h = value_h_stride;
    kv.params[1].output_stride_b = output_value_b_stride;
    kv.params[1].output_stride_m = output_value_m_stride;
    kv.params[1].output_stride_h = output_value_h_stride;
    kv.params[1].count = count;
    kv.params[1].output_khm = output_value_k * output_value_h * output_value_m;
    kv.params[1].output_kh = output_value_k * output_value_h;
    kv.params[1].output_k = output_value_k;

    constexpr uint32_t block_size = 256;
    const dim3 grid_size((count - 1 + block_size) / block_size, 2);

    const int64_t elem_size = size_of_data_type * pack_size;
    cudaStream_t cuda_stream = ctx->stream()->As<ep::CudaStream>()->cuda_stream();
    if (elem_size == 16) {
      BatchConcatPastKeyValue<16, int32_t><<<grid_size, block_size, 0, cuda_stream>>>(kv);
    } else if (elem_size == 8) {
      BatchConcatPastKeyValue<8, int32_t><<<grid_size, block_size, 0, cuda_stream>>>(kv);
    } else if (elem_size == 4) {
      BatchConcatPastKeyValue<4, int32_t><<<grid_size, block_size, 0, cuda_stream>>>(kv);
    } else if (elem_size == 2) {
      BatchConcatPastKeyValue<2, int32_t><<<grid_size, block_size, 0, cuda_stream>>>(kv);
    } else if (elem_size == 1) {
      BatchConcatPastKeyValue<1, int32_t><<<grid_size, block_size, 0, cuda_stream>>>(kv);
    } else {
      UNIMPLEMENTED();
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("fused_attention_concat_past_key_value")
    .SetCreateFn<FusedAttentionConcatPastKeyValueKernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA));

template<typename T, typename PositionType, typename IndexType, size_t num_dims,
         size_t rotary_emb_dim>
struct FusedApplyRotaryEmbParam {
  const T* x;
  const T* cos;
  const T* sin;
  const PositionType* position_ids;
  T* out;
  const T theta;
  const float inv_actual_rotary_size;  // 1.0 / (rotary_size per rotary dimension)
  const IndexType actual_rotary_size;  // rotary_size per rotary dimension
  const IndexType rotary_size;
  const IndexType rotate_stride;
  const IndexType k0;
  const IndexType k1;
  IndexType num_elements;
  const IndexType k;
  const IndexType x_offset;

  IndexType ref_stride[num_dims];  // b, m, h, k
  IndexType out_stride[num_dims];  // ordered descendingly by stride
  IndexType x_stride[num_dims];

  IndexType position_b_stride;
  IndexType position_rotate_stride;

  IndexType sinuous_m_stride;

  FusedApplyRotaryEmbParam(const T* x, const T* cos, const T* sin, const PositionType* position_ids,
                           T* out, const T theta, const float inv_actual_rotary_size,
                           const IndexType actual_rotary_size, const IndexType rotary_size,
                           const IndexType rotate_stride, const IndexType num_elements,
                           const IndexType k, const IndexType k0, const IndexType k1,
                           const IndexType x_offset)
      : x(x),
        cos(cos),
        sin(sin),
        position_ids(position_ids),
        out(out),
        theta(theta),
        inv_actual_rotary_size(inv_actual_rotary_size),
        actual_rotary_size(actual_rotary_size),
        rotary_size(rotary_size),
        rotate_stride(rotate_stride),
        num_elements(num_elements),
        k(k),
        k0(k0),
        k1(k1),
        x_offset(x_offset) {}
};

template<typename T, typename PositionType, typename IndexType, size_t PackSize, size_t num_dims,
         size_t rotary_emb_dim>
__global__ void IntervalKernel(
    FusedApplyRotaryEmbParam<T, PositionType, IndexType, num_dims, rotary_emb_dim> param) {
  for (IndexType packed_offset = threadIdx.x + blockIdx.x * blockDim.x;
       packed_offset < param.num_elements; packed_offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, PackSize>;
    IndexType offset = packed_offset * PackSize;
    IndexType index[num_dims];  // b, m, h, k

    IndexType temp_offset = offset;

    for (int i = 0; i < num_dims - 1; i++) {
      IndexType ref_stride = param.ref_stride[i];
      IndexType idx = temp_offset / ref_stride;
      index[i] = idx;
      temp_offset = temp_offset - idx * ref_stride;
    }
    index[num_dims - 1] = temp_offset;

    IndexType x_offset = param.x_offset;
    IndexType out_offset = 0;
#pragma unroll
    for (int i = 0; i < num_dims; i++) {
      x_offset = x_offset + param.x_stride[i] * index[i];
      out_offset = out_offset + param.out_stride[i] * index[i];
    }
    const LoadPack x_vec = *reinterpret_cast<const LoadPack*>(param.x + x_offset);

    const IndexType k_index = index[num_dims - 1];
    if (k_index < param.rotary_size) {
      const IndexType position_rotate_index = (k_index >= param.k0) ? 1 : 0;
      const IndexType b_index = index[0], m_index = index[1];
      const IndexType position_id_offset = b_index * param.position_b_stride
                                           + position_rotate_index * param.position_rotate_stride
                                           + m_index;

      const PositionType position =
          param.position_ids ? param.position_ids[position_id_offset] : m_index;
      const IndexType actual_k_index = k_index % param.actual_rotary_size;
      const IndexType sinuous_offset = position * param.sinuous_m_stride + actual_k_index;

      LoadPack cos_vec, sin_vec, out_vec;

      if (param.cos && param.sin) {
        cos_vec = *reinterpret_cast<const LoadPack*>(param.cos + sinuous_offset);
        sin_vec = *reinterpret_cast<const LoadPack*>(param.sin + sinuous_offset);
      } else {
        const IndexType actual_ndim = param.rotary_size / rotary_emb_dim;
#pragma unroll
        for (int i = 0; i < PackSize / 2; i++) {
          T val = position
                  * expf(2.0f * static_cast<float>(((actual_k_index >> 1) + i))
                         * param.inv_actual_rotary_size * logf(param.theta));
          T cos_val = cosf(val);
          T sin_val = sinf(val);
          cos_vec.elem[i * 2] = cos_val;
          cos_vec.elem[i * 2 + 1] = cos_val;
          sin_vec.elem[i * 2] = sin_val;
          sin_vec.elem[i * 2 + 1] = sin_val;
        }
      }

#pragma unroll
      for (int i = 0; i < PackSize / 2; i++) {
        out_vec.elem[i * 2] =
            x_vec.elem[i * 2] * cos_vec.elem[i * 2] - x_vec.elem[i * 2 + 1] * sin_vec.elem[i * 2];
        out_vec.elem[i * 2 + 1] = x_vec.elem[i * 2 + 1] * cos_vec.elem[i * 2 + 1]
                                  + x_vec.elem[i * 2] * sin_vec.elem[i * 2 + 1];
      }

      *(reinterpret_cast<LoadPack*>(param.out + out_offset)) = out_vec;
    } else {
      *(reinterpret_cast<LoadPack*>(param.out + out_offset)) = x_vec;
    }
  }
}

template<typename T, typename PositionType, typename IndexType, size_t num_dims,
         size_t rotary_emb_dim>
__global__ void PlaneKernel(
    FusedApplyRotaryEmbParam<T, PositionType, IndexType, num_dims, rotary_emb_dim> param) {
  for (IndexType offset = threadIdx.x + blockIdx.x * blockDim.x; offset < param.num_elements;
       offset += blockDim.x * gridDim.x) {
    using LoadPack = cuda::elementwise::Packed<T, 2>;
    IndexType temp_offset = offset;
    IndexType index[num_dims];
#pragma unroll
    for (int i = 0; i < num_dims - 1; i++) {
      IndexType ref_stride = param.ref_stride[i];
      IndexType idx = temp_offset / ref_stride;
      index[i] = idx;
      temp_offset = temp_offset - idx * ref_stride;
    }
    index[num_dims - 1] = temp_offset;

    const IndexType b_index = index[0], m_index = index[1], k_index = index[num_dims - 1];
    const IndexType position_rotate_index = (k_index >= param.k0) ? 1 : 0;
    const IndexType position_id_offset = b_index * param.position_b_stride
                                         + position_rotate_index * param.position_rotate_stride
                                         + m_index;

    const PositionType position =
        param.position_ids ? param.position_ids[position_id_offset] : m_index;
    const IndexType actual_k_index = k_index % param.actual_rotary_size;
    const IndexType sinuous_offset = position * param.k + actual_k_index;

    T cos_val, sin_val, out_val;

    if (param.cos && param.sin) {
      cos_val = *(param.cos + sinuous_offset);
      sin_val = *(param.sin + sinuous_offset);
    } else {
      T val = position
              * expf(2.0f * static_cast<float>(k_index % (param.actual_rotary_size >> 1))
                     * param.inv_actual_rotary_size * logf(param.theta));
      cos_val = cosf(val);
      sin_val = sinf(val);
    }

    LoadPack x_vec;
    IndexType x_offset = param.x_offset;
    IndexType out_offset = 0;
#pragma unroll
    for (int i = 0; i < num_dims; i++) {
      x_offset = x_offset + param.x_stride[i] * index[i];
      out_offset = out_offset + param.out_stride[i] * index[i];
    }

    if (k_index < param.k0) {
      x_vec.elem[0] = *(param.x + x_offset);
      x_vec.elem[1] = (param.k0 - k_index > param.rotate_stride)
                          ? static_cast<T>(-*(param.x + x_offset + param.rotate_stride))
                          : *(param.x + x_offset - param.rotate_stride);
      out_val = cos_val * x_vec.elem[0] + sin_val * x_vec.elem[1];
    } else if (k_index < param.k1) {
      x_vec.elem[0] = *(param.x + x_offset);
      x_vec.elem[1] = (param.k1 - k_index > param.rotate_stride)
                          ? static_cast<T>(-*(param.x + x_offset + param.rotate_stride))
                          : *(param.x + x_offset - param.rotate_stride);
      out_val = cos_val * x_vec.elem[0] + sin_val * x_vec.elem[1];
    } else {
      out_val = *(param.x + x_offset);
    }

    *(param.out + out_offset) = out_val;
  }
}

template<typename T, typename PositionType, typename IndexType, size_t PackSize, size_t num_dims,
         size_t rotary_emb_dim>
void LaunchKernel(ep::CudaStream* stream, const T* x, const T* cos, const T* sin,
                  const PositionType* position_ids, T* out, const int64_t* position_shape,
                  const std::string& x_layout, const std::string& output_layout,
                  const std::string& mode, const T theta, const IndexType rotary_size,
                  const IndexType b, const IndexType m, const IndexType h, const IndexType k,
                  const IndexType x_b_stride, const IndexType x_m_stride,
                  const IndexType x_h_stride, const IndexType x_offset,
                  const IndexType out_b_stride, const IndexType out_m_stride,
                  const IndexType out_h_stride, IndexType num_elements) {
  const IndexType k0 = rotary_size / rotary_emb_dim,
                  k1 = rotary_size;  // TODO: this only support 1d, 2d, rotary postional encoding

  const IndexType rotate_stride = rotary_size / (2 * rotary_emb_dim);

  const IndexType actual_rotary_size = rotary_size / rotary_emb_dim;
  const float inv_actual_rotary_size = 1.0 / actual_rotary_size;

  struct FusedApplyRotaryEmbParam<T, PositionType, IndexType, num_dims, rotary_emb_dim> param(
      x, cos, sin, position_ids, out, theta, inv_actual_rotary_size, actual_rotary_size,
      rotary_size, rotate_stride, num_elements, k, k0, k1, x_offset);

  const IndexType ref_strides[num_dims] = {m * h * k, h * k, k, 1};
  const IndexType out_strides[num_dims] = {out_b_stride, out_m_stride, out_h_stride, 1};
  const IndexType x_strides[num_dims] = {x_b_stride, x_m_stride, x_h_stride, 1};

  param.sinuous_m_stride = actual_rotary_size;

  const IndexType position_m = position_shape ? static_cast<IndexType>(position_shape[2]) : m;
  param.position_rotate_stride = position_m;
  param.position_b_stride = position_m * rotary_emb_dim;

// K has to be the last dimension, only k&m matters, therefore strides other than k&m does not
// really needs to be computed
#pragma unroll
  for (int i = 0; i < num_dims; i++) {
    param.ref_stride[i] = ref_strides[i];
    param.out_stride[i] = out_strides[i];
    param.x_stride[i] = x_strides[i];
  }

  constexpr size_t blk_size = 128;

  if (mode == "plane") {
    param.num_elements = param.num_elements * PackSize;
    PlaneKernel<T, PositionType, IndexType, num_dims, rotary_emb_dim>
        <<<(param.num_elements + blk_size - 1) / blk_size, blk_size, 0, stream->cuda_stream()>>>(
            param);
  } else {
    IntervalKernel<T, PositionType, IndexType, PackSize, num_dims, rotary_emb_dim>
        <<<(param.num_elements + blk_size - 1) / blk_size, blk_size, 0, stream->cuda_stream()>>>(
            param);
  }
}

template<typename T, typename PositionType, typename IndexType, size_t num_dims,
         size_t rotary_emb_dim>
void DispatchPackSize(ep::CudaStream* stream, const T* x, const T* cos, const T* sin,
                      const PositionType* position_ids, T* out, const int64_t* position_shape,
                      const std::string& x_layout, const std::string& output_layout,
                      const std::string& mode, const T theta, const IndexType rotary_size,
                      const IndexType b, const IndexType m, const IndexType h, const IndexType k,
                      const IndexType x_b_stride, const IndexType x_m_stride,
                      const IndexType x_h_stride, const IndexType x_offset,
                      const IndexType out_b_stride, const IndexType out_m_stride,
                      const IndexType out_h_stride, IndexType num_elements) {
  const auto CheckPackSize = [&](const size_t PackSize) {
    bool r = (((reinterpret_cast<uintptr_t>(x) % (sizeof(T) * PackSize)) == 0)
              && (((rotary_size / rotary_emb_dim) % PackSize) == 0)
              && (((k - rotary_size) % PackSize) == 0) && ((16 / sizeof(T)) >= PackSize));
    return r;
  };

  if (CheckPackSize(8)) {
    num_elements /= 8;
    LaunchKernel<T, PositionType, IndexType, 8, num_dims, rotary_emb_dim>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode,
        theta, rotary_size, b, m, h, k, x_b_stride, x_m_stride, x_h_stride, x_offset, out_b_stride,
        out_m_stride, out_h_stride, num_elements);
  } else if (CheckPackSize(4)) {
    num_elements /= 4;
    LaunchKernel<T, PositionType, IndexType, 4, num_dims, rotary_emb_dim>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode,
        theta, rotary_size, b, m, h, k, x_b_stride, x_m_stride, x_h_stride, x_offset, out_b_stride,
        out_m_stride, out_h_stride, num_elements);
  } else {
    num_elements /= 2;
    LaunchKernel<T, PositionType, IndexType, 2, num_dims, rotary_emb_dim>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode,
        theta, rotary_size, b, m, h, k, x_b_stride, x_m_stride, x_h_stride, x_offset, out_b_stride,
        out_m_stride, out_h_stride, num_elements);
  }
}

template<typename T, typename PositionType, size_t num_dims, size_t rotary_emb_dim>
void DispatchIndex(ep::CudaStream* stream, const T* x, const T* cos, const T* sin,
                   const PositionType* position_ids, T* out, const int64_t* position_shape,
                   const std::string& x_layout, const std::string& output_layout,
                   const std::string& mode, const T theta, const int64_t rotary_size,
                   const int64_t b, const int64_t m, const int64_t h, const int64_t k,
                   const int64_t x_b_stride, const int64_t x_m_stride, const int64_t x_h_stride,
                   const int64_t x_offset, const int64_t out_b_stride, const int64_t out_m_stride,
                   const int64_t out_h_stride) {
  int64_t num_elements = b * m * h * k;
  if (num_elements < (1 << 30)) {
    DispatchPackSize<T, PositionType, int32_t, num_dims, rotary_emb_dim>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode,
        theta, static_cast<int32_t>(rotary_size), static_cast<int32_t>(b), static_cast<int32_t>(m),
        static_cast<int32_t>(h), static_cast<int32_t>(k), static_cast<int32_t>(x_b_stride),
        static_cast<int32_t>(x_m_stride), static_cast<int32_t>(x_h_stride),
        static_cast<int32_t>(x_offset), static_cast<int32_t>(out_b_stride),
        static_cast<int32_t>(out_m_stride), static_cast<int32_t>(out_h_stride),
        static_cast<int32_t>(num_elements));
  } else {
    DispatchPackSize<T, PositionType, int64_t, num_dims, rotary_emb_dim>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode,
        theta, rotary_size, b, m, h, k, x_b_stride, x_m_stride, x_h_stride, x_offset, out_b_stride,
        out_m_stride, out_h_stride, num_elements);
  }
}

template<typename T, typename PositionType, size_t num_dims>
void DispatchRotaryEmbeddingDimension(ep::CudaStream* stream, const T* x, const T* cos,
                                      const T* sin, const PositionType* position_ids, T* out,
                                      const int64_t* position_shape, const std::string& x_layout,
                                      const std::string& output_layout, const std::string& mode,
                                      const T theta, const int64_t rotary_size,
                                      const int rotary_emb_dim, const int64_t b, const int64_t m,
                                      const int64_t h, const int64_t k, const int64_t x_b_stride,
                                      const int64_t x_m_stride, const int64_t x_h_stride,
                                      const int64_t x_offset, const int64_t out_b_stride,
                                      const int64_t out_m_stride, const int64_t out_h_stride) {
  if (rotary_emb_dim == 1) {
    DispatchIndex<T, PositionType, num_dims, 1>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode,
        theta, rotary_size, b, m, h, k, x_b_stride, x_m_stride, x_h_stride, x_offset, out_b_stride,
        out_m_stride, out_h_stride);
  } else if (rotary_emb_dim == 2) {
    DispatchIndex<T, PositionType, num_dims, 2>(
        stream, x, cos, sin, position_ids, out, position_shape, x_layout, output_layout, mode,
        theta, rotary_size, b, m, h, k, x_b_stride, x_m_stride, x_h_stride, x_offset, out_b_stride,
        out_m_stride, out_h_stride);
  }
}

template<typename T, typename PositionType>
class FusedApplyRotaryEmbKernel final : public user_op::OpKernel {
 public:
  FusedApplyRotaryEmbKernel() = default;
  ~FusedApplyRotaryEmbKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* cos = nullptr;
    user_op::Tensor* sin = nullptr;
    user_op::Tensor* position_ids = nullptr;
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const std::string& x_layout = ctx->Attr<std::string>("x_layout");
    const std::string& output_layout = ctx->Attr<std::string>("output_layout");
    const std::string& mode = ctx->Attr<std::string>("mode");
    const int64_t tensor_index = ctx->Attr<int64_t>("tensor_index");
    const int64_t k_size = ctx->Attr<int64_t>("k_size");
    const int64_t rotary_size = ctx->Attr<int64_t>("rotary_size");
    const float theta = 1.0f / ctx->Attr<float>("base");
    int rotary_emb_dim = 1;

    if (ctx->has_input("cos", 0)) { cos = ctx->Tensor4ArgNameAndIndex("cos", 0); }

    if (ctx->has_input("sin", 0)) { sin = ctx->Tensor4ArgNameAndIndex("sin", 0); }

    if (ctx->has_input("position_ids", 0)) {
      position_ids = ctx->Tensor4ArgNameAndIndex("position_ids", 0);
      rotary_emb_dim = position_ids->shape_view().At(1);
    }

    constexpr size_t ndims = 4;
    int64_t b = 0;
    int64_t m = 0;
    int64_t h = 0;
    int64_t k = 0;
    int64_t out_b_stride = 0, out_m_stride = 0, out_h_stride = 0, out_offset = 0;
    int64_t x_b_stride = 0, x_m_stride = 0, x_h_stride = 0, x_offset = 0;

    ParseDims(out->shape_view(), output_layout, Optional<int64_t>(), k_size, 0, &b, &m, &h, &k,
              &out_b_stride, &out_m_stride, &out_h_stride, &out_offset);
    ParseDims(x->shape_view(), x_layout, Optional<int64_t>(), k_size, tensor_index, &b, &m, &h, &k,
              &x_b_stride, &x_m_stride, &x_h_stride, &x_offset);

    // TODO: hard code num_dims & seems redundant template problem...
    DispatchRotaryEmbeddingDimension<T, PositionType, ndims>(
        ctx->stream()->As<ep::CudaStream>(), reinterpret_cast<const T*>(x->dptr()),
        cos ? reinterpret_cast<const T*>(cos->dptr()) : nullptr,
        sin ? reinterpret_cast<const T*>(sin->dptr()) : nullptr,
        position_ids ? reinterpret_cast<const PositionType*>(position_ids->dptr()) : nullptr,
        reinterpret_cast<T*>(out->mut_dptr()),
        position_ids ? position_ids->shape_view().data() : nullptr, x_layout, output_layout, mode,
        static_cast<T>(theta), rotary_size, rotary_emb_dim, b, m, h, k, x_b_stride, x_m_stride,
        x_h_stride, x_offset, out_b_stride, out_m_stride, out_h_stride);
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_APPLY_ROTARY_EMB_GPU(dtype, position_type)          \
  REGISTER_USER_KERNEL("fused_apply_rotary_emb")                           \
      .SetCreateFn<FusedApplyRotaryEmbKernel<dtype, position_type>>()      \
      .SetIsMatchedHob(                                                    \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                  \
          && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value) \
          && (user_op::HobInputSize("position_ids") == 1)                  \
          && (user_op::HobDataType("position_ids", 0) == GetDataType<position_type>::value));

#define REGISTER_FUSED_APPLY_ROTARY_EMB_GPU_DTYPE(dtype)                                \
  REGISTER_FUSED_APPLY_ROTARY_EMB_GPU(dtype, int64_t);                                  \
  REGISTER_FUSED_APPLY_ROTARY_EMB_GPU(dtype, int32_t);                                  \
  REGISTER_USER_KERNEL("fused_apply_rotary_emb")                                        \
      .SetCreateFn<FusedApplyRotaryEmbKernel<dtype, int64_t>>()                         \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                  \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobInputSize("position_ids") == 0));

REGISTER_FUSED_APPLY_ROTARY_EMB_GPU_DTYPE(float);
REGISTER_FUSED_APPLY_ROTARY_EMB_GPU_DTYPE(half);
#if CUDA_VERSION >= 11000
REGISTER_FUSED_APPLY_ROTARY_EMB_GPU_DTYPE(nv_bfloat16);
#endif  // CUDA_VERSION >= 11000

}  // namespace

}  // namespace user_op

}  // namespace oneflow

#endif  // WITH_CUTLASS
