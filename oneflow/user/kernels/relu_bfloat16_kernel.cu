#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include <hip/hip_runtime.h>
#include "oneflow/core/ep/cuda/hip_stream.h"

#if defined(CUDA_VERSION) && CUDA_VERSION >= 11000

#include "oneflow/core/device/cuda_pseudo_bfloat16.h"

namespace oneflow {

namespace user_op {

namespace {

template<typename T>
__global__ void ReluBackwardGpu(int64_t n, const T* y, const T* dy, T* dx) {
  const T zero = static_cast<T>(0.0);
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = y[i] > zero ? dy[i] : zero; }
}

}  // namespace

class ReluGradNvBFloat16Kernel final : public OpKernel {
 public:
  ReluGradNvBFloat16Kernel() = default;
  ~ReluGradNvBFloat16Kernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(KernelComputeContext* ctx) const override {
    const Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const int64_t n = y->shape_view().elem_cnt();
    ReluBackwardGpu<hip_bfloat16><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                                   ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        n, reinterpret_cast<const hip_bfloat16*>(y->dptr()),
        reinterpret_cast<const hip_bfloat16*>(dy->dptr()),
        reinterpret_cast<hip_bfloat16*>(dx->mut_dptr()));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("relu_grad")
    .SetCreateFn<ReluGradNvBFloat16Kernel>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)
                     && (user_op::HobDataType("dx", 0) == DataType::kBFloat16))
    .SetInplaceProposalFn([](const user_op::InferContext&,
                             user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> {
      OF_RETURN_IF_ERROR(AddInplaceArgPairFn("dx", 0, "dy", 0, true));
      return Maybe<void>::Ok();
    });

}  // namespace user_op

}  // namespace oneflow

#endif  // defined(CUDA_VERSION) && CUDA_VERSION >= 11000
