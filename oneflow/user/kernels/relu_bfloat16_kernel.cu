#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include <hip/hip_runtime.h>

#if defined(CUDA_VERSION) && CUDA_VERSION >= 11000

#include "oneflow/core/device/cuda_pseudo_bfloat16.h"

namespace oneflow {

namespace user_op {

namespace {

template<typename T>
__global__ void ReluForwardGpu(int64_t n, const T* in, T* out) {
  const T zero = static_cast<T>(0.0);
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T in_i = in[i];
    T out_i = zero;
    if (in_i > zero) { out_i = in_i; }
    out[i] = out_i;
  }
}

template<typename T>
__global__ void ReluBackwardGpu(int64_t n, const T* y, const T* dy, T* dx) {
  const T zero = static_cast<T>(0.0);
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = y[i] > zero ? dy[i] : zero; }
}

}  // namespace

class ReluNvBFloat16Kernel final : public OpKernel {
 public:
  ReluNvBFloat16Kernel() = default;
  ~ReluNvBFloat16Kernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(KernelComputeContext* ctx) const override {
    const Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int64_t n = in->shape().elem_cnt();
    ReluForwardGpu<hip_bfloat16>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->device_ctx()->hip_stream()>>>(
            n, reinterpret_cast<const hip_bfloat16*>(in->dptr()),
            reinterpret_cast<hip_bfloat16*>(out->mut_dptr()));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

class ReluGradNvBFloat16Kernel final : public OpKernel {
 public:
  ReluGradNvBFloat16Kernel() = default;
  ~ReluGradNvBFloat16Kernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(KernelComputeContext* ctx) const override {
    const Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const int64_t n = y->shape().elem_cnt();
    ReluBackwardGpu<hip_bfloat16>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->device_ctx()->hip_stream()>>>(
            n, reinterpret_cast<const hip_bfloat16*>(y->dptr()),
            reinterpret_cast<const hip_bfloat16*>(dy->dptr()),
            reinterpret_cast<hip_bfloat16*>(dx->mut_dptr()));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("relu")
    .SetCreateFn<ReluNvBFloat16Kernel>()
    .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")
                     & (user_op::HobDataType("out", 0) == DataType::kBFloat16))
    .SetInplaceProposalFn([](const user_op::InferContext&,
                             user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> {
      OF_RETURN_IF_ERROR(AddInplaceArgPairFn("out", 0, "in", 0, true));
      return Maybe<void>::Ok();
    });

REGISTER_USER_KERNEL("relu_grad")
    .SetCreateFn<ReluGradNvBFloat16Kernel>()
    .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")
                     & (user_op::HobDataType("dx", 0) == DataType::kBFloat16))
    .SetInplaceProposalFn([](const user_op::InferContext&,
                             user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> {
      OF_RETURN_IF_ERROR(AddInplaceArgPairFn("dx", 0, "dy", 0, true));
      return Maybe<void>::Ok();
    });

}  // namespace user_op

}  // namespace oneflow

#endif  // defined(CUDA_VERSION) && CUDA_VERSION >= 11000
