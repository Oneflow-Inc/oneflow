#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ndarray/ndarray_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/ep/cuda/primitive/unary_functor.cuh"
#include "oneflow/core/cuda/layer_norm.cuh"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/kernel/cuda_graph_support.h"

#ifdef WITH_CUTLASS
#include <cutlass/fast_math.h>
#endif  // WITH_CUTLASS

namespace oneflow {

namespace {

template<typename SRC, typename DST, ep::primitive::UnaryOp activation, bool affine>
struct AffineStore {
  AffineStore(DST* y, int64_t row_size, int64_t channel_size, int64_t spatial_size,
              const DST* gamma, const DST* beta)
      : y(y),
        row_size(row_size),
        channel_size(channel_size),
        spatial_size(spatial_size),
        gamma(gamma),
        beta(beta),
        act(0, 0) {}

  template<int PackSize>
  __device__ void store(const SRC* src, int64_t row, int64_t col) {
    cuda::layer_norm::Pack<DST, PackSize> y_pack;
    const int64_t offset = row * row_size + col;
    const int64_t packed_offset = offset / PackSize;
    const int64_t gamma_beta_offset = (offset / spatial_size) % channel_size;
    DST gamma_val = 1.0;
    DST beta_val = 0.0;
    if (affine) {
      gamma_val = gamma[gamma_beta_offset];
      beta_val = beta[gamma_beta_offset];
    }

#pragma unroll
    for (int i = 0; i < PackSize; ++i) {
      DST normalized_i = static_cast<DST>(src[i]);
      if (affine) {
        y_pack.elem[i] = act(normalized_i * gamma_val + beta_val);
      } else {
        // Direct Store.
        y_pack.elem[i] = act(normalized_i);
      }
    }
    *(reinterpret_cast<cuda::layer_norm::PackType<DST, PackSize>*>(y) + packed_offset) =
        y_pack.storage;
  }
  bool CanPackAs(size_t pack_size) { return (spatial_size % pack_size) == 0; }
  DST* y;
  int64_t row_size;
  int64_t channel_size;
  int64_t spatial_size;
  const DST* gamma;
  const DST* beta;
  ep::primitive::UnaryFunctor<DeviceType::kCUDA, activation, DST, DST> act;
};

template<typename SRC, typename DST, bool affine>
struct ScaleLoad {
  using LoadType = DST;
  ScaleLoad(const SRC* src, const SRC* gamma, int64_t row_size, int64_t channel_size,
            int64_t spatial_size)
      : src(src),
        gamma(gamma),
        row_size(row_size),
        channel_size(channel_size),
        spatial_size(spatial_size) {}
  template<int PackSize>
  __device__ void load(DST* dst, int64_t row, int64_t col) const {
    cuda::layer_norm::Pack<SRC, PackSize> src_pack;
    cuda::layer_norm::Pack<SRC, PackSize> gamma_pack;

    const int64_t offset = row * row_size + col;
    const int64_t packed_offset = offset / PackSize;
    const int64_t gamma_offset = (offset / spatial_size) % channel_size;

    src_pack.storage =
        *(reinterpret_cast<const cuda::layer_norm::PackType<SRC, PackSize>*>(src) + packed_offset);
    SRC gamma_val = static_cast<SRC>(1.0);
    if (affine) { gamma_val = gamma[gamma_offset]; }
#pragma unroll
    for (int i = 0; i < PackSize; ++i) { dst[i] = static_cast<DST>(src_pack.elem[i] * gamma_val); }
  }
  bool CanPackAs(size_t pack_size) { return (spatial_size % pack_size) == 0; }
  const SRC* src;
  const SRC* gamma;
  int64_t row_size;
  int64_t channel_size;
  int64_t spatial_size;
};

#ifdef WITH_CUTLASS

template<typename SRC, typename DST, ep::primitive::UnaryOp activation, bool affine>
struct ChannelsLastStore {
  ChannelsLastStore(DST* y, const DST* gamma, const DST* beta, int64_t spatial_size,
                    int64_t channel_size, int64_t num_groups)
      : y(y),
        gamma(gamma),
        beta(beta),
        spatial_size(spatial_size),
        c0(num_groups),
        c1(channel_size / num_groups),
        act(0, 0) {}

  template<int PackSize>
  __device__ void store(const SRC* src, int32_t row, int32_t col) {
    cuda::layer_norm::Pack<DST, PackSize> y_pack;
    cuda::layer_norm::Pack<DST, PackSize> gamma_pack;
    cuda::layer_norm::Pack<DST, PackSize> beta_pack;
    int32_t spatial_idx;
    int32_t c1_idx;
    c1(spatial_idx, c1_idx, col);
    int32_t batch_idx;
    int32_t c0_idx;
    c0(batch_idx, c0_idx, row);
    const int32_t y_offset =
        (batch_idx * c0.divisor * c1.divisor * spatial_size + spatial_idx * c0.divisor * c1.divisor
         + c0_idx * c1.divisor + c1_idx)
        / PackSize;
    const int32_t gamma_beta_offset = (c0_idx * c1.divisor + c1_idx) / PackSize;
    if (affine) {
      gamma_pack.storage =
          *(reinterpret_cast<const cuda::layer_norm::PackType<DST, PackSize>*>(gamma)
            + gamma_beta_offset);
      beta_pack.storage = *(reinterpret_cast<const cuda::layer_norm::PackType<DST, PackSize>*>(beta)
                            + gamma_beta_offset);
    }

#pragma unroll
    for (int i = 0; i < PackSize; ++i) {
      DST normalized_i = static_cast<DST>(src[i]);
      if (affine) {
        y_pack.elem[i] = act(normalized_i * gamma_pack.elem[i] + beta_pack.elem[i]);
      } else {
        // Direct Store.
        y_pack.elem[i] = act(normalized_i);
      }
    }
    *(reinterpret_cast<cuda::layer_norm::PackType<DST, PackSize>*>(y) + y_offset) = y_pack.storage;
  }
  bool CanPackAs(size_t pack_size) { return (c1.divisor % pack_size) == 0; }
  DST* y;
  const DST* gamma;
  const DST* beta;
  int32_t spatial_size;
  cutlass::FastDivmod c0;
  cutlass::FastDivmod c1;
  ep::primitive::UnaryFunctor<DeviceType::kCUDA, activation, DST, DST> act;
};

template<typename SRC, typename DST>
struct ChannelsLastLoad {
  using LoadType = DST;
  ChannelsLastLoad(const SRC* src, int64_t spatial_size, int64_t channel_size, int64_t num_groups)
      : src(src), spatial_size(spatial_size), c0(num_groups), c1(channel_size / num_groups) {}
  template<int N>
  __device__ void load(DST* dst, int32_t row, int32_t col) const {
    int32_t spatial_idx;
    int32_t c1_idx;
    c1(spatial_idx, c1_idx, col);
    int32_t batch_idx;
    int32_t c0_idx;
    c0(batch_idx, c0_idx, row);
    cuda::layer_norm::Pack<SRC, N> pack;
    const int32_t offset = (batch_idx * c0.divisor * c1.divisor * spatial_size
                            + spatial_idx * c0.divisor * c1.divisor + c0_idx * c1.divisor + c1_idx)
                           / N;

    pack.storage = *(reinterpret_cast<const cuda::layer_norm::PackType<SRC, N>*>(src) + offset);
#pragma unroll
    for (int i = 0; i < N; ++i) { dst[i] = static_cast<DST>(pack.elem[i]); }
  }
  bool CanPackAs(size_t pack_size) { return (c1.divisor % pack_size) == 0; }
  const SRC* src;
  int32_t spatial_size;
  cutlass::FastDivmod c0;
  cutlass::FastDivmod c1;
};

#else

template<typename SRC, typename DST, ep::primitive::UnaryOp activation, bool affine>
struct ChannelsLastStore {
  ChannelsLastStore(DST* y, const DST* gamma, const DST* beta, int64_t spatial_size,
                    int64_t channel_size, int64_t num_groups)
      : y(y),
        gamma(gamma),
        beta(beta),
        spatial_size(spatial_size),
        c0(num_groups),
        c1(channel_size / num_groups),
        act(0, 0) {}

  template<int PackSize>
  __device__ void store(const SRC* src, int32_t row, int32_t col) {
    cuda::layer_norm::Pack<DST, PackSize> y_pack;
    cuda::layer_norm::Pack<DST, PackSize> gamma_pack;
    cuda::layer_norm::Pack<DST, PackSize> beta_pack;
    int32_t spatial_idx = col / c1;
    int32_t c1_idx = col - spatial_idx * c1;
    int32_t batch_idx = row / c0;
    int32_t c0_idx = row - batch_idx * c0;
    const int32_t y_offset =
        (batch_idx * c0 * c1 * spatial_size + spatial_idx * c0 * c1 + c0_idx * c1 + c1_idx)
        / PackSize;
    const int32_t gamma_beta_offset = (c0_idx * c1 + c1_idx) / PackSize;
    if (affine) {
      gamma_pack.storage =
          *(reinterpret_cast<const cuda::layer_norm::PackType<DST, PackSize>*>(gamma)
            + gamma_beta_offset);
      beta_pack.storage = *(reinterpret_cast<const cuda::layer_norm::PackType<DST, PackSize>*>(beta)
                            + gamma_beta_offset);
    }

#pragma unroll
    for (int i = 0; i < PackSize; ++i) {
      DST normalized_i = static_cast<DST>(src[i]);
      if (affine) {
        y_pack.elem[i] = act(normalized_i * gamma_pack.elem[i] + beta_pack.elem[i]);
      } else {
        // Direct Store.
        y_pack.elem[i] = act(normalized_i);
      }
    }
    *(reinterpret_cast<cuda::layer_norm::PackType<DST, PackSize>*>(y) + y_offset) = y_pack.storage;
  }
  bool CanPackAs(size_t pack_size) { return (c1 % pack_size) == 0; }
  DST* y;
  const DST* gamma;
  const DST* beta;
  int32_t spatial_size;
  int32_t c0;
  int32_t c1;
  ep::primitive::UnaryFunctor<DeviceType::kCUDA, activation, DST, DST> act;
};

template<typename SRC, typename DST>
struct ChannelsLastLoad {
  using LoadType = DST;
  ChannelsLastLoad(const SRC* src, int64_t spatial_size, int64_t channel_size, int64_t num_groups)
      : src(src), spatial_size(spatial_size), c0(num_groups), c1(channel_size / num_groups) {}
  template<int N>
  __device__ void load(DST* dst, int32_t row, int32_t col) const {
    int32_t spatial_idx = col / c1;
    int32_t c1_idx = col - spatial_idx * c1;
    int32_t batch_idx = row / c0;
    int32_t c0_idx = row - batch_idx * c0;
    cuda::layer_norm::Pack<SRC, N> pack;
    const int32_t offset =
        (batch_idx * c0 * c1 * spatial_size + spatial_idx * c0 * c1 + c0_idx * c1 + c1_idx) / N;

    pack.storage = *(reinterpret_cast<const cuda::layer_norm::PackType<SRC, N>*>(src) + offset);
#pragma unroll
    for (int i = 0; i < N; ++i) { dst[i] = static_cast<DST>(pack.elem[i]); }
  }
  bool CanPackAs(size_t pack_size) { return (c1 % pack_size) == 0; }
  const SRC* src;
  int32_t spatial_size;
  int32_t c0;
  int32_t c1;
};

#endif  // WITH_CUTLASS

template<typename T, ep::primitive::UnaryOp activation, bool affine>
void GroupNormForwardGpu(ep::Stream* stream, const int64_t num_instances, const int64_t norm_size,
                         const int64_t channel_size, const int64_t spatial_size,
                         const double epsilon, const T* x_ptr, const T* gamma_ptr,
                         const T* beta_ptr, T* y_ptr, user_op::Tensor* mean,
                         user_op::Tensor* inv_variance, bool channels_first) {
  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  if (channels_first) {
    cuda::layer_norm::DirectLoad<T, T> load(x_ptr, norm_size);
    AffineStore<ComputeType, T, activation, affine> store(y_ptr, norm_size, channel_size,
                                                          spatial_size, gamma_ptr, beta_ptr);

    cuda::layer_norm::DispatchLayerNorm<decltype(load), decltype(store), ComputeType>(
        stream->As<ep::CudaStream>()->hip_stream(), load, store, num_instances, norm_size, epsilon,
        mean->mut_dptr<ComputeType>(), inv_variance->mut_dptr<ComputeType>());
  } else {
    ChannelsLastLoad<T, T> load(x_ptr, spatial_size, channel_size,
                                channel_size / (norm_size / spatial_size));
    ChannelsLastStore<ComputeType, T, activation, affine> store(
        y_ptr, gamma_ptr, beta_ptr, spatial_size, channel_size,
        channel_size / (norm_size / spatial_size));

    cuda::layer_norm::DispatchLayerNorm<decltype(load), decltype(store), ComputeType>(
        stream->As<ep::CudaStream>()->hip_stream(), load, store, num_instances, norm_size, epsilon,
        mean->mut_dptr<ComputeType>(), inv_variance->mut_dptr<ComputeType>());
  }
}

template<typename T, ep::primitive::UnaryOp activation>
void DispatchGroupNormAffine(ep::Stream* stream, const int64_t num_instances,
                             const int64_t norm_size, const int64_t channel_size,
                             const int64_t spatial_size, const double epsilon, const T* x_ptr,
                             const T* gamma_ptr, const T* beta_ptr, T* y_ptr, user_op::Tensor* mean,
                             user_op::Tensor* inv_variance, bool channels_first) {
  if (gamma_ptr != nullptr && beta_ptr != nullptr) {
    GroupNormForwardGpu<T, activation, true>(stream, num_instances, norm_size, channel_size,
                                             spatial_size, epsilon, x_ptr, gamma_ptr, beta_ptr,
                                             y_ptr, mean, inv_variance, channels_first);
  } else {
    GroupNormForwardGpu<T, activation, false>(stream, num_instances, norm_size, channel_size,
                                              spatial_size, epsilon, x_ptr, gamma_ptr, beta_ptr,
                                              y_ptr, mean, inv_variance, channels_first);
  }
}

template<typename T>
void DispatchGroupNormForwardGpu(ep::Stream* stream, const int64_t num_instances,
                                 const int64_t norm_size, const int64_t channel_size,
                                 const int64_t spatial_size, const double epsilon, const T* x_ptr,
                                 const T* gamma_ptr, const T* beta_ptr, T* y_ptr,
                                 user_op::Tensor* mean, user_op::Tensor* inv_variance,
                                 bool channels_first, const std::string& activation) {
  if (activation == "none") {
    DispatchGroupNormAffine<T, ep::primitive::UnaryOp::kIdentity>(
        stream, num_instances, norm_size, channel_size, spatial_size, epsilon, x_ptr, gamma_ptr,
        beta_ptr, y_ptr, mean, inv_variance, channels_first);
  } else if (activation == "silu") {
    DispatchGroupNormAffine<T, ep::primitive::UnaryOp::kSilu>(
        stream, num_instances, norm_size, channel_size, spatial_size, epsilon, x_ptr, gamma_ptr,
        beta_ptr, y_ptr, mean, inv_variance, channels_first);
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T, bool affine>
void GroupNormBackwardGpu(ep::Stream* stream, const int64_t num_instances, const int64_t norm_size,
                          const int64_t channel_size, const int64_t spatial_size, const T* dy_ptr,
                          const T* x_ptr, const user_op::Tensor* mean,
                          const user_op::Tensor* inv_variance, const T* gamma_ptr, T* dx_ptr) {
  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  cuda::layer_norm::DirectLoad<T, T> load_x(x_ptr, norm_size);
  ScaleLoad<T, T, affine> load_scaled_dy(dy_ptr, gamma_ptr, norm_size, channel_size, spatial_size);
  cuda::layer_norm::DirectStore<ComputeType, T> store(dx_ptr, norm_size);
  OF_CUDA_CHECK((cuda::layer_norm::DispatchLayerNormGrad<decltype(load_x), decltype(load_scaled_dy),
                                                         decltype(store), ComputeType>(
      stream->As<ep::CudaStream>()->hip_stream(), load_x, load_scaled_dy, store,
      mean->dptr<ComputeType>(), inv_variance->dptr<ComputeType>(), num_instances, norm_size)));
}

template<typename T>
void LaunchGroupNormBackward(ep::Stream* stream, const int64_t num_instances,
                             const int64_t norm_size, const int64_t channel_size,
                             const int64_t spatial_size, const T* dy_ptr, const T* x_ptr,
                             const user_op::Tensor* mean, const user_op::Tensor* inv_variance,
                             const T* gamma_ptr, T* dx_ptr) {
  if (gamma_ptr != nullptr) {
    GroupNormBackwardGpu<T, true>(stream, num_instances, norm_size, channel_size, spatial_size,
                                  dy_ptr, x_ptr, mean, inv_variance, gamma_ptr, dx_ptr);
  } else {
    GroupNormBackwardGpu<T, false>(stream, num_instances, norm_size, channel_size, spatial_size,
                                   dy_ptr, x_ptr, mean, inv_variance, gamma_ptr, dx_ptr);
  }
}

}  // namespace

template<typename T>
class GroupNormGpuKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  GroupNormGpuKernel() = default;
  ~GroupNormGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    const double epsilon = ctx->Attr<double>("epsilon");
    const int32_t num_groups = ctx->Attr<int32_t>("num_groups");
    const std::string& data_format = ctx->Attr<std::string>("data_format");
    CHECK_GE(epsilon, HIPDNN_BN_MIN_EPSILON);
    const int64_t num_instances = mean->shape_view().elem_cnt();  // N*num_groups
    const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
    const int64_t batch_size = x->shape_view().At(0);
    int64_t channel_size = 0;
    bool channels_first = false;
    if (data_format == "channels_first") {
      channel_size = x->shape_view().At(1);
      channels_first = true;
    } else if (data_format == "channels_last") {
      channel_size = x->shape_view().At(x->shape_view().NumAxes() - 1);
      channels_first = false;
    } else {
      UNIMPLEMENTED();
    }
    const int64_t spatial_size = x->shape_view().elem_cnt() / batch_size / channel_size;
    const T* gamma_ptr = nullptr;
    const T* beta_ptr = nullptr;
    if (ctx->has_input("gamma", 0) && ctx->has_input("beta", 0)) {
      const user_op::Tensor* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
      gamma_ptr = gamma->dptr<T>();
      CHECK_EQ(gamma->shape_view().elem_cnt(), channel_size);
      const user_op::Tensor* beta = ctx->Tensor4ArgNameAndIndex("beta", 0);
      beta_ptr = ctx->Tensor4ArgNameAndIndex("beta", 0)->dptr<T>();
      CHECK_EQ(beta->shape_view().elem_cnt(), channel_size);
    }
    DispatchGroupNormForwardGpu<T>(ctx->stream(), num_instances, norm_size, channel_size,
                                   spatial_size, epsilon, x->dptr<T>(), gamma_ptr, beta_ptr,
                                   y->mut_dptr<T>(), mean, inv_variance, channels_first,
                                   ctx->Attr<std::string>("activation"));
  }
};

#define REGISTER_GROUP_NORM_CUDA_KERNEL(dtype)                         \
  REGISTER_USER_KERNEL("group_norm")                                   \
      .SetCreateFn<GroupNormGpuKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value));

REGISTER_GROUP_NORM_CUDA_KERNEL(half)
REGISTER_GROUP_NORM_CUDA_KERNEL(float)
REGISTER_GROUP_NORM_CUDA_KERNEL(double)
#if CUDA_VRSION >= 11000
REGISTER_GROUP_NORM_CUDA_KERNEL(hip_bfloat16)
#endif

template<typename T>
class GroupNormGradGpuKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  GroupNormGradGpuKernel() = default;
  ~GroupNormGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const int64_t num_instances = mean->shape_view().elem_cnt();
    const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
    const int64_t batch_size = x->shape_view().At(0);
    const int64_t channel_size = x->shape_view().At(1);
    const int64_t spatial_size = x->shape_view().elem_cnt() / batch_size / channel_size;
    const T* gamma_ptr = nullptr;
    if (ctx->has_input("gamma", 0)) {
      gamma_ptr = ctx->Tensor4ArgNameAndIndex("gamma", 0)->dptr<T>();
    }
    LaunchGroupNormBackward<T>(ctx->stream(), num_instances, norm_size, channel_size, spatial_size,
                               dy->dptr<T>(), x->dptr<T>(), mean, inv_variance, gamma_ptr,
                               dx->mut_dptr<T>());
  };
};

#define REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(dtype)                    \
  REGISTER_USER_KERNEL("group_norm_grad")                              \
      .SetCreateFn<GroupNormGradGpuKernel<dtype>>()                    \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value));

REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(half)
REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(float)
REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(double)
#if CUDA_VRSION >= 11000
REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(hip_bfloat16)
#endif

constexpr int kReduceBlockSize = 512;
constexpr int kBlockSize = 128;
constexpr int kNumWaves = 32;

inline hipError_t GetReduceNumBlocks(int64_t n, int* num_blocks) {
  int dev;
  {
    hipError_t err = hipGetDevice(&dev);
    if (err != hipSuccess) { return err; }
  }
  int sm_count;
  {
    hipError_t err = hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev);
    if (err != hipSuccess) { return err; }
  }
  int tpm;
  {
    hipError_t err = hipDeviceGetAttribute(&tpm, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
    if (err != hipSuccess) { return err; }
  }
  *num_blocks =
      std::max<int>(1, std::min<int64_t>(n, sm_count * tpm / kReduceBlockSize * kNumWaves));
  return hipSuccess;
}

inline hipError_t GetNumBlocks(int64_t n, int* num_blocks) {
  int dev;
  {
    hipError_t err = hipGetDevice(&dev);
    if (err != hipSuccess) { return err; }
  }
  int sm_count;
  {
    hipError_t err = hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev);
    if (err != hipSuccess) { return err; }
  }
  int tpm;
  {
    hipError_t err = hipDeviceGetAttribute(&tpm, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
    if (err != hipSuccess) { return err; }
  }
  *num_blocks = std::max<int>(1, std::min<int64_t>((n + kBlockSize - 1) / kBlockSize,
                                                   sm_count * tpm / kBlockSize * kNumWaves));
  return hipSuccess;
}

template<typename T>
struct SumOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const { return a + b; }
};

template<typename T, int PackSize>
struct GetPackType {
  using type = typename std::aligned_storage<sizeof(T) * PackSize, sizeof(T) * PackSize>::type;
};

template<typename T, int PackSize>
using PackType = typename GetPackType<T, PackSize>::type;

template<typename T, int PackSize>
union Pack {
  static_assert(sizeof(PackType<T, PackSize>) == sizeof(T) * PackSize, "");
  __device__ Pack(T val) {
    for (int i = 0; i < PackSize; i++) { elem[i] = val; }
  }

  T elem[PackSize];
  PackType<T, PackSize> storage;
};

constexpr int kMaxPackBytes = 128 / 8;
constexpr int kMaxPackSize = 8;

constexpr int Min(int a, int b) { return a < b ? a : b; }

template<typename T>
constexpr int GetPackSize() {
  return Min(kMaxPackBytes / sizeof(T), kMaxPackSize);
}

template<typename T, typename ComputeType, int PackSize>
__global__ void GroupNormParamGradKernel(const T* dy, const T* x, const ComputeType* mean,
                                         const ComputeType* inv_var,
                                         ComputeType* dgamma_partial_sum,
                                         ComputeType* dbeta_partial_sum, const int32_t batch_size,
                                         const int32_t group_size, const int32_t channel_size,
                                         const int32_t spatial_size) {
  using LoadType = PackType<T, PackSize>;
  const int32_t batch_channel_size = batch_size * channel_size;
  for (int32_t batch_channel_id = blockIdx.x; batch_channel_id < batch_channel_size;
       batch_channel_id += gridDim.x) {
    const int32_t batch_id = batch_channel_id / channel_size;
    const int32_t channel_id = batch_channel_id % channel_size;
    const int32_t group_num = channel_size / group_size;
    const int32_t batch_group_id = batch_id * group_size + channel_id / group_num;

    ComputeType mean_val = mean[batch_group_id];
    ComputeType inv_var_val = inv_var[batch_group_id];

    Pack<ComputeType, PackSize> ds_sum_pack(0);
    Pack<ComputeType, PackSize> db_sum_pack(0);

    for (int32_t spatial = threadIdx.x * PackSize; spatial < spatial_size;
         spatial += blockDim.x * PackSize) {
      Pack<T, PackSize> dy_pack(0);
      Pack<T, PackSize> x_pack(0);
      const int32_t load_idx = batch_channel_id * spatial_size + spatial;
      const LoadType* dy_load = reinterpret_cast<const LoadType*>(dy + load_idx);
      dy_pack.storage = *dy_load;
      const LoadType* x_load = reinterpret_cast<const LoadType*>(x + load_idx);
      x_pack.storage = *x_load;
#pragma unroll
      for (int i = 0; i < PackSize; i++) {
        ds_sum_pack.elem[i] += static_cast<ComputeType>(dy_pack.elem[i])
                               * (static_cast<ComputeType>(x_pack.elem[i]) - mean_val)
                               * inv_var_val;
        db_sum_pack.elem[i] += static_cast<ComputeType>(dy_pack.elem[i]);
      }
    }

    ComputeType ds_sum = 0.0;
    ComputeType db_sum = 0.0;

#pragma unroll
    for (int i = 0; i < PackSize; i++) {
      ds_sum += ds_sum_pack.elem[i];
      db_sum += db_sum_pack.elem[i];
    }

    __syncthreads();
    typedef hipcub::BlockReduce<ComputeType, kReduceBlockSize> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage1;
    __shared__ typename BlockReduce::TempStorage temp_storage2;
    ComputeType ds_sum_result = BlockReduce(temp_storage1).Reduce(ds_sum, SumOp<ComputeType>());
    ComputeType db_sum_result = BlockReduce(temp_storage2).Reduce(db_sum, SumOp<ComputeType>());
    if (threadIdx.x == 0) {
      dgamma_partial_sum[batch_channel_id] = ds_sum_result;
      dbeta_partial_sum[batch_channel_id] = db_sum_result;
    }
  }
}

template<typename T, typename ComputeType>
__global__ void BatchReduceGammaBetaGradKernel(ComputeType* ds_sum, ComputeType* db_sum, T* dgamma,
                                               T* dbeta, const int32_t batch_size,
                                               const int32_t group_size, const int32_t channel_size,
                                               const int32_t spatial_size) {
  const int32_t group_num = channel_size / group_size;
  CUDA_1D_KERNEL_LOOP(channel_idx, channel_size) {
    ComputeType dgamma_sum = 0.0;
    ComputeType dbeta_sum = 0.0;
    for (int batch_id = 0; batch_id < batch_size; batch_id++) {
      const int32_t batch_group_id = batch_id * group_size + channel_idx / group_num;
      const int32_t batch_channel_id = batch_id * channel_size + channel_idx;
      dgamma_sum += ds_sum[batch_channel_id];
      dbeta_sum += db_sum[batch_channel_id];
    }
    dgamma[channel_idx] = dgamma_sum;
    dbeta[channel_idx] = dbeta_sum;
  }
}

template<typename T>
int32_t GetLaunchPackSize(const int32_t spatial_size) {
  for (int pack_size = GetPackSize<T>(); pack_size > 0; pack_size /= 2) {
    if (spatial_size % pack_size == 0) { return pack_size; }
  }
  return 1;
}

template<typename T, typename ComputeType>
void DispatchGroupNormParamGradKernel(ep::Stream* stream, const T* dy, const T* x,
                                      const ComputeType* mean, const ComputeType* inv_var,
                                      ComputeType* reduce_ds_buf, ComputeType* reduce_db_buf,
                                      const int32_t batch_size, const int32_t group_size,
                                      const int32_t channel_size, const int32_t spatial_size) {
  const int launch_pack_size = GetLaunchPackSize<T>(spatial_size);
  int num_blocks;
  OF_CUDA_CHECK(GetReduceNumBlocks(batch_size * channel_size, &num_blocks));
  if (launch_pack_size == 8) {
    GroupNormParamGradKernel<T, ComputeType, 8>
        <<<num_blocks, kReduceBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            dy, x, mean, inv_var, reduce_ds_buf, reduce_db_buf, batch_size, group_size,
            channel_size, spatial_size);
  } else if (launch_pack_size == 4) {
    GroupNormParamGradKernel<T, ComputeType, 4>
        <<<num_blocks, kReduceBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            dy, x, mean, inv_var, reduce_ds_buf, reduce_db_buf, batch_size, group_size,
            channel_size, spatial_size);
  } else if (launch_pack_size == 2) {
    GroupNormParamGradKernel<T, ComputeType, 2>
        <<<num_blocks, kReduceBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            dy, x, mean, inv_var, reduce_ds_buf, reduce_db_buf, batch_size, group_size,
            channel_size, spatial_size);
  } else {
    GroupNormParamGradKernel<T, ComputeType, 1>
        <<<num_blocks, kReduceBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            dy, x, mean, inv_var, reduce_ds_buf, reduce_db_buf, batch_size, group_size,
            channel_size, spatial_size);
  }
}

template<typename T>
class GroupNormParamGradGpuKernel final : public user_op::OpKernel,
                                          public user_op::CudaGraphSupport {
 public:
  GroupNormParamGradGpuKernel() = default;
  ~GroupNormParamGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    user_op::Tensor* dgamma = ctx->Tensor4ArgNameAndIndex("dgamma", 0);
    user_op::Tensor* dbeta = ctx->Tensor4ArgNameAndIndex("dbeta", 0);
    const int64_t num_instances = mean->shape_view().elem_cnt();
    const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
    const int64_t batch_size = x->shape_view().At(0);
    const int64_t channel_size = x->shape_view().At(1);
    const int64_t spatial_size = x->shape_view().elem_cnt() / batch_size / channel_size;
    const int64_t group_size = num_instances / batch_size;
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
    ComputeType* reduce_ds_buf_ptr = reinterpret_cast<ComputeType*>(tmp_buffer->mut_dptr<char>());
    ComputeType* reduce_db_buf_ptr = reinterpret_cast<ComputeType*>(
        tmp_buffer->mut_dptr<char>() + batch_size * channel_size * sizeof(T));
    DispatchGroupNormParamGradKernel<T, ComputeType>(
        ctx->stream(), dy->dptr<T>(), x->dptr<T>(), mean->dptr<ComputeType>(),
        inv_variance->dptr<ComputeType>(), reduce_ds_buf_ptr, reduce_db_buf_ptr, batch_size,
        group_size, channel_size, spatial_size);
    int num_blocks;
    OF_CUDA_CHECK(GetNumBlocks(channel_size, &num_blocks));
    // Note(zhengzekang): In large batchsize, it is recommend to use gemm to reduce. (1, N) matmul
    // (N, C)
    BatchReduceGammaBetaGradKernel<T, ComputeType>
        <<<num_blocks, kBlockSize, 0, ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
            reduce_ds_buf_ptr, reduce_db_buf_ptr, dgamma->mut_dptr<T>(), dbeta->mut_dptr<T>(),
            batch_size, group_size, channel_size, spatial_size);
  };
};

#define REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(dtype, compute_dtype)                  \
  REGISTER_USER_KERNEL("group_norm_param_grad")                                           \
      .SetCreateFn<GroupNormParamGradGpuKernel<dtype>>()                                  \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                 \
        const auto& x = ctx->InputTensorDesc("x", 0);                                     \
        const int64_t batch_size = x.shape().At(0);                                       \
        const int64_t channel_size = x.shape().At(1);                                     \
        size_t tmp_buffer_size = (2 * batch_size * channel_size) * sizeof(compute_dtype); \
        return tmp_buffer_size;                                                           \
      })                                                                                  \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                    \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value));

REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(half, float)
REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(float, float)
REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(double, double)
#if CUDA_VRSION >= 11000
REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(hip_bfloat16, float)
#endif

}  // namespace oneflow
