#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/scalar_logical_kernels.h"
#include "oneflow/user/kernels/elementwise_xpu_kernel.cuh"

namespace oneflow {

template<template<typename T> class BIN_OP, typename T>
__global__ void DoCUDAScalarLogical(const int64_t elem_cnt, const T scalar, const T* in,
                                    int8_t* out) {
  DoScalarLogical<BIN_OP, T>(elem_cnt, scalar, in, out);
}

template<template<typename T> class BIN_OP, typename T>
struct ScalarLogicalFunctor<DeviceType::kGPU, BIN_OP, T> final {
  void operator()(DeviceCtx* ctx, const int64_t elem_cnt, const T scalar, const T* in,
                  int8_t* out) {
    RUN_CUDA_KERNEL((DoCUDAScalarLogical<BIN_OP, T>), ctx, BlocksNum4ThreadsNum(elem_cnt), elem_cnt,
                    scalar, in, out);
  }
};

INSTANTIATE_SCALAR_LOGICAL_FUNCTORS(DeviceType::kGPU, BinaryFuncEQ);
INSTANTIATE_SCALAR_LOGICAL_FUNCTORS(DeviceType::kGPU, BinaryFuncNE);
INSTANTIATE_SCALAR_LOGICAL_FUNCTORS(DeviceType::kGPU, BinaryFuncGT);
INSTANTIATE_SCALAR_LOGICAL_FUNCTORS(DeviceType::kGPU, BinaryFuncGE);
INSTANTIATE_SCALAR_LOGICAL_FUNCTORS(DeviceType::kGPU, BinaryFuncLT);
INSTANTIATE_SCALAR_LOGICAL_FUNCTORS(DeviceType::kGPU, BinaryFuncLE);

}  // namespace oneflow
