/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/device/nccl_util.h"
#include "oneflow/core/job/eager_nccl_comm_manager.h"
#include "oneflow/core/job/parallel_desc.h"

namespace oneflow {

class EagerNcclAllReduceKernel final : public user_op::OpKernel {
 public:
  EagerNcclAllReduceKernel() = default;
  ~EagerNcclAllReduceKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    CHECK_EQ(in->shape(), out->shape());
    CHECK_EQ(in->data_type(), out->data_type());
    std::set<std::pair<int64_t, int64_t>> device_set;
    const std::string& parallel_conf_txt = ctx->Attr<std::string>("parallel_conf");
    ParallelConf parallel_conf{};
    CHECK(TxtString2PbMessage(parallel_conf_txt, &parallel_conf));
    const ParallelDesc parallel_desc(parallel_conf);
    FOR_RANGE(int64_t, parallel_id, 0, parallel_desc.parallel_num()) {
      int64_t machine_id = CHECK_JUST(parallel_desc.MachineId4ParallelId(parallel_id));
      int64_t device_id = CHECK_JUST(parallel_desc.DeviceId4ParallelId(parallel_id));
      device_set.emplace(std::make_pair(machine_id, device_id));
    }
    ncclComm_t comm = CHECK_NOTNULL(Global<EagerNcclCommMgr>::Get())->GetCommForDevice(device_set);
    OF_NCCL_CHECK(ncclAllReduce(in->dptr(), out->mut_dptr(), in->shape().elem_cnt(),
                                GetNcclDataType(in->data_type()), ncclSum, comm,
                                ctx->device_ctx()->hip_stream()));
  };
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("eager_nccl_all_reduce")
    .SetCreateFn<EagerNcclAllReduceKernel>()
    .SetIsMatchedHob(user_op::HobDeviceTag() == "gpu");

}  // namespace oneflow
