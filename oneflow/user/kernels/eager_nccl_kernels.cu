#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/common/container_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/device/nccl_util.h"
#include "oneflow/core/job/eager_nccl_comm_manager.h"
#include "oneflow/core/job/parallel_desc.h"

namespace oneflow {

namespace {

class EagerNcclOpKernelState final : public user_op::OpKernelState {
 public:
  EagerNcclOpKernelState(user_op::KernelInitContext* ctx) { Init(ctx); }
  ~EagerNcclOpKernelState() override = default;

  Symbol<ParallelDesc> parallel_desc() const { return parallel_desc_; }
  ncclComm_t comm() const { return comm_; }

 private:
  void Init(user_op::KernelInitContext* ctx) {
    const std::string& parallel_conf_txt = ctx->Attr<std::string>("parallel_conf");
    ParallelConf parallel_conf;
    std::set<std::pair<int64_t, int64_t>> device_set;
    CHECK(TxtString2PbMessage(parallel_conf_txt, &parallel_conf));
    parallel_desc_ = SymbolOf(ParallelDesc(parallel_conf));
    FOR_RANGE(int64_t, parallel_id, 0, parallel_desc_->parallel_num()) {
      int64_t machine_id = CHECK_JUST(parallel_desc_->MachineId4ParallelId(parallel_id));
      int64_t device_id = CHECK_JUST(parallel_desc_->DeviceId4ParallelId(parallel_id));
      device_set.emplace(std::make_pair(machine_id, device_id));
    }
    comm_ = CHECK_NOTNULL(Global<EagerNcclCommMgr>::Get())->GetCommForDevice(device_set);
  }

  Symbol<ParallelDesc> parallel_desc_;
  ncclComm_t comm_;
};

}  // namespace

class EagerNcclAllReduceKernel final : public user_op::OpKernel {
 public:
  EagerNcclAllReduceKernel() = default;
  ~EagerNcclAllReduceKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EagerNcclOpKernelState>(ctx);
  }

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    auto* kernel_state = dynamic_cast<EagerNcclOpKernelState*>(state);
    CHECK(kernel_state != nullptr);
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    CHECK_EQ(in->shape(), out->shape());
    CHECK_EQ(in->data_type(), out->data_type());
    OF_NCCL_CHECK(ncclAllReduce(in->dptr(), out->mut_dptr(), in->shape().elem_cnt(),
                                GetNcclDataType(in->data_type()), ncclSum, kernel_state->comm(),
                                ctx->device_ctx()->hip_stream()));
  };
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("eager_nccl_all_reduce")
    .SetCreateFn<EagerNcclAllReduceKernel>()
    .SetIsMatchedHob(user_op::HobDeviceTag() == "gpu");

class EagerNcclBroadcastKernel final : public user_op::OpKernel {
 public:
  EagerNcclBroadcastKernel() = default;
  ~EagerNcclBroadcastKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EagerNcclOpKernelState>(ctx);
  }

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    auto* kernel_state = dynamic_cast<EagerNcclOpKernelState*>(state);
    CHECK(kernel_state != nullptr);
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    CHECK_EQ(in->shape(), out->shape());
    CHECK_EQ(in->data_type(), out->data_type());
    int64_t root = ctx->Attr<int64_t>("root");
    OF_NCCL_CHECK(ncclBroadcast(in->dptr(), out->mut_dptr(), in->shape().elem_cnt(),
                                GetNcclDataType(in->data_type()), root, kernel_state->comm(),
                                ctx->device_ctx()->hip_stream()));
  };
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("eager_nccl_broadcast")
    .SetCreateFn<EagerNcclBroadcastKernel>()
    .SetIsMatchedHob(user_op::HobDeviceTag() == "gpu");

class EagerNcclReduceKernel final : public user_op::OpKernel {
 public:
  EagerNcclReduceKernel() = default;
  ~EagerNcclReduceKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EagerNcclOpKernelState>(ctx);
  }

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    auto* kernel_state = dynamic_cast<EagerNcclOpKernelState*>(state);
    CHECK(kernel_state != nullptr);
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    CHECK_EQ(in->shape(), out->shape());
    CHECK_EQ(in->data_type(), out->data_type());
    int64_t root = ctx->Attr<int64_t>("root");
    OF_NCCL_CHECK(ncclReduce(in->dptr(), out->mut_dptr(), in->shape().elem_cnt(),
                             GetNcclDataType(in->data_type()), ncclSum, root, kernel_state->comm(),
                             ctx->device_ctx()->hip_stream()));
  };
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("eager_nccl_reduce")
    .SetCreateFn<EagerNcclReduceKernel>()
    .SetIsMatchedHob(user_op::HobDeviceTag() == "gpu");

class EagerNcclReduceScatterKernel final : public user_op::OpKernel {
 public:
  EagerNcclReduceScatterKernel() = default;
  ~EagerNcclReduceScatterKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EagerNcclOpKernelState>(ctx);
  }

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    auto* kernel_state = dynamic_cast<EagerNcclOpKernelState*>(state);
    CHECK(kernel_state != nullptr);
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    CHECK(!(in->shape() == out->shape()));
    CHECK_EQ(in->data_type(), out->data_type());
    const auto& op_type = ctx->Attr<std::string>("op_type");
    OF_NCCL_CHECK(ncclReduceScatter(in->dptr(), out->mut_dptr(), out->shape().elem_cnt(),
                                    GetNcclDataType(in->data_type()),
                                    CHECK_JUST(MapAt(op_type2ncclRedOp_t, op_type)),
                                    kernel_state->comm(), ctx->device_ctx()->hip_stream()));
  };
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }

  static HashMap<std::string, ncclRedOp_t> op_type2ncclRedOp_t;
};

HashMap<std::string, ncclRedOp_t> EagerNcclReduceScatterKernel::op_type2ncclRedOp_t = {
    {"sum", ncclSum}, {"max", ncclMax}};

REGISTER_USER_KERNEL("eager_nccl_reduce_scatter")
    .SetCreateFn<EagerNcclReduceScatterKernel>()
    .SetIsMatchedHob(user_op::HobDeviceTag() == "gpu");

class EagerNcclAllGatherKernel final : public user_op::OpKernel {
 public:
  EagerNcclAllGatherKernel() = default;
  ~EagerNcclAllGatherKernel() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<EagerNcclOpKernelState>(ctx);
  }

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    auto* kernel_state = dynamic_cast<EagerNcclOpKernelState*>(state);
    CHECK(kernel_state != nullptr);
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    CHECK(!(in->shape() == out->shape()));
    CHECK_EQ(in->data_type(), out->data_type());
    OF_NCCL_CHECK(ncclAllGather(in->dptr(), out->mut_dptr(), in->shape().elem_cnt(),
                                GetNcclDataType(in->data_type()), kernel_state->comm(),
                                ctx->device_ctx()->hip_stream()));
  };
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("eager_nccl_all_gather")
    .SetCreateFn<EagerNcclAllGatherKernel>()
    .SetIsMatchedHob(user_op::HobDeviceTag() == "gpu");
}  // namespace oneflow
