/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/user/kernels/cublas_fused_mlp_util.cuh"
#include "oneflow/core/ep/include/primitive/memcpy.h"
#include "oneflow/core/ep/cuda/cuda_device.h"
// HIPBLASLT_EPILOGUE_BGRADB only support in cuda11.4.2 or higher version.
// TODO(zhengzekang): In cuda11.6 version, HIPBLASLT_EPILOGUE_BGRADB may occur illegal memory access
// error in some shapes.
#if CUDA_VERSION >= 11060

namespace oneflow {

namespace {

hipDataType GetGemmComputeType(hipDataType data_type) {
  switch (data_type) {
    case HIP_R_32F: return HIP_R_32F;
    case HIP_R_64F: return HIP_R_64F;
    case HIP_R_16F: return HIP_R_32F;
#if CUDA_VERSION >= 11000
    case HIP_R_16BF: return HIP_R_32F;
#endif  // CUDA_VERSION >= 11000
    default: UNIMPLEMENTED(); return HIP_R_32F;
  }
}

template<typename T>
class CublasMatmulBiasAddGradKernel final : public user_op::OpKernel,
                                            public user_op::CudaGraphSupport {
 public:
  CublasMatmulBiasAddGradKernel() = default;
  ~CublasMatmulBiasAddGradKernel() override = default;

  std::shared_ptr<user_op::OpKernelCache> InitOpKernelCache(
      user_op::KernelCacheContext* ctx) const override {
    return CreateCublasFusedMLPKernelCache();
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState*,
               const user_op::OpKernelCache* cache) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* w_grad = ctx->Tensor4ArgNameAndIndex("w_grad", 0);
    user_op::Tensor* b_grad = ctx->Tensor4ArgNameAndIndex("b_grad", 0);
    const auto* matmul_grad_cache =
        CHECK_NOTNULL(dynamic_cast<const CublasFusedMLPKernelCache*>(cache));
    auto* hip_stream = ctx->stream()->As<ep::CudaStream>();

    const DataType data_type = dy->data_type();
    const hipblasComputeType_t cublas_compute_dtype = GetComputeType(data_type);
    const hipDataType cuda_data_type = GetCudaDataType(data_type);
    size_t cublas_m = 0, cublas_n = 0, cublas_k = 0;
    int64_t cublas_lda = 0, cublas_ldb = 0, cublas_ldc = 0;
    const double alpha = 1.0;
    const auto sp_alpha = GetCublasScalarParameter(alpha, cublas_compute_dtype);
    const double beta = 0.0;
    const auto sp_beta = GetCublasScalarParameter(beta, cublas_compute_dtype);

    // currently only support 2D matmul.
    DimVector dy_shape(2);
    dy->shape_view().ToDimVector(&dy_shape);
    DimVector x_shape(2);
    x->shape_view().ToDimVector(&x_shape);
    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_BGRADB;

    InferMatmulCublasMNK(dy_shape, x_shape,
                         /*transpose_a=*/ep::primitive::BlasTransposeType::T,
                         /*transpose_b=*/ep::primitive::BlasTransposeType::N, &cublas_m, &cublas_n,
                         &cublas_k, &cublas_lda, &cublas_ldb, &cublas_ldc);
    if (cublas_k != 1) {
      SetCublasAttr(
          matmul_grad_cache, cublas_compute_dtype, cuda_data_type, /*need_aux=*/false,
          /*transpose_a=*/ep::primitive::BlasTransposeType::T,
          /*transpose_b=*/ep::primitive::BlasTransposeType::N, epilogue, b_grad->mut_dptr(),
          /*aux_ptr=*/nullptr, cublas_m, cublas_n, cublas_k, cublas_lda, cublas_ldb, cublas_ldc);

      /*
      a = dy, b = x
      cublas_a=x, cublas_b=dy
      */
      OF_CUBLAS_CHECK(hipblasLtMatmul(
          hip_stream->cublas_lt_handle(), matmul_grad_cache->operation_desc, &sp_alpha, x->dptr(),
          matmul_grad_cache->cublas_a_desc, dy->dptr(), matmul_grad_cache->cublas_b_desc, &sp_beta,
          w_grad->mut_dptr(), matmul_grad_cache->cublas_c_desc, w_grad->mut_dptr(),
          matmul_grad_cache->cublas_c_desc, nullptr, hip_stream->cublas_workspace(),
          hip_stream->cublas_workspace_size(), hip_stream->hip_stream()));
    } else {
// Cause cublasLtmatmul get wrong bias grad in cublas_k == 1.
#if CUDA_VERSION >= 11000
      hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;
#else
      hipblasGemmAlgo_t algo =
          (data_type == DataType::kFloat16) ? CUBLAS_GEMM_DFALT_TENSOR_OP : HIPBLAS_GEMM_DEFAULT;
#endif

      hipDataType gemm_compute_type = GetGemmComputeType(cuda_data_type);
      std::unique_ptr<ep::primitive::Memcpy> memcpy_primitive =
          ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(
              ctx->stream()->device_type(), ep::primitive::MemcpyKind::kDtoD);
      CHECK(memcpy_primitive);
      memcpy_primitive->Launch(ctx->stream(), b_grad->mut_dptr(), dy->dptr(), cublas_n * sizeof(T));
      OF_CUBLAS_CHECK(hipblasGemmEx(
          hip_stream->cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, cublas_m, cublas_n, cublas_k,
          &sp_alpha, x->dptr(), cuda_data_type, cublas_lda, dy->dptr(), cuda_data_type, cublas_ldb,
          &sp_beta, w_grad->mut_dptr(), cuda_data_type, cublas_ldc, gemm_compute_type, algo));
    }
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUBLAS_MATMUL_BIAS_ADD_GRAD_KERNEL(dtype)             \
  REGISTER_USER_KERNEL("cublas_matmul_bias_add_grad")                  \
      .SetCreateFn<CublasMatmulBiasAddGradKernel<dtype>>()             \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value));

REGISTER_CUBLAS_MATMUL_BIAS_ADD_GRAD_KERNEL(float)
REGISTER_CUBLAS_MATMUL_BIAS_ADD_GRAD_KERNEL(double)
REGISTER_CUBLAS_MATMUL_BIAS_ADD_GRAD_KERNEL(half)

}  // namespace

}  // namespace oneflow

#endif  // CUDA_VERSION >= 11060
