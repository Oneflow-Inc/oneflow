#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/unique_kernel_util.h"
#include "oneflow/core/cuda/unique.cuh"
#include "oneflow/core/device/nccl_util.h"
#include "oneflow/core/job/eager_nccl_comm_manager.h"
#include "oneflow/core/job/parallel_desc.h"
#include "oneflow/core/ep/include/primitive/memcpy.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/user/kernels/gather_kernel_util.h"
#include "oneflow/user/kernels/unsorted_segment_sum_kernel_util.h"
#include <hipcub/hipcub.hpp>

namespace oneflow {

namespace {

void DumpToFile(ep::Stream* stream, std::string filename, int64_t parallel_id, size_t data_size,
                const void* ptr) {
  void* host_ptr;
  OF_CUDA_CHECK(hipHostMalloc(&host_ptr, data_size));
  std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
      ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                ep::primitive::MemcpyKind::kDtoH);
  CHECK(copyd2h_primitive);
  copyd2h_primitive->Launch(stream, host_ptr, ptr, data_size);
  CHECK_JUST(stream->Sync());
  std::ofstream dx_os;
  dx_os.open(StrCat("test/" + filename + "_", parallel_id));
  dx_os.write(reinterpret_cast<char*>(host_ptr), data_size);
  dx_os.close();
  OF_CUDA_CHECK(hipHostFree(host_ptr));
}

template<typename T>
void DebugEmbeddingShuffle(user_op::KernelComputeContext* ctx) {
  const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
  user_op::Tensor* cur_rank_embeddings = ctx->Tensor4ArgNameAndIndex("cur_rank_embeddings", 0);
  user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
  DumpToFile(ctx->stream(), "cur_rank_embeddings", parallel_id,
             cur_rank_embeddings->shape().elem_cnt() * sizeof(T), cur_rank_embeddings->dptr());
  DumpToFile(ctx->stream(), "embeddings", parallel_id, embeddings->shape().elem_cnt() * sizeof(T),
             embeddings->dptr());
}

template<typename T>
void DebugEmbeddingGradientShuffle(user_op::KernelComputeContext* ctx) {
  const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
  user_op::Tensor* cur_rank_unique_embedding_diff =
      ctx->Tensor4ArgNameAndIndex("cur_rank_unique_embedding_diff", 0);
  user_op::Tensor* embedding_diff = ctx->Tensor4ArgNameAndIndex("embedding_diff", 0);
  DumpToFile(ctx->stream(), "cur_rank_unique_embedding_diff", parallel_id,
             cur_rank_unique_embedding_diff->shape().elem_cnt() * sizeof(T),
             cur_rank_unique_embedding_diff->dptr());
  DumpToFile(ctx->stream(), "embedding_diff", parallel_id,
             embedding_diff->shape().elem_cnt() * sizeof(T), embedding_diff->dptr());
}

template<typename K, typename IDX>
void DebugIdShuffle(user_op::KernelComputeContext* ctx) {
  const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
  const user_op::Tensor* ids = ctx->Tensor4ArgNameAndIndex("ids", 0);
  user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
  user_op::Tensor* ids_reverse_idx = ctx->Tensor4ArgNameAndIndex("ids_reverse_idx", 0);
  user_op::Tensor* cur_rank_num_unique_ids =
      ctx->Tensor4ArgNameAndIndex("cur_rank_num_unique_ids", 0);
  user_op::Tensor* cur_rank_unique_ids = ctx->Tensor4ArgNameAndIndex("cur_rank_unique_ids", 0);
  user_op::Tensor* cur_rank_reverse_idx = ctx->Tensor4ArgNameAndIndex("cur_rank_reverse_idx", 0);
  user_op::Tensor* num_unique_ids_matrix = ctx->Tensor4ArgNameAndIndex("num_unique_ids_matrix", 0);
  DumpToFile(ctx->stream(), "ids", parallel_id, ids->shape().elem_cnt() * sizeof(K), ids->dptr());
  DumpToFile(ctx->stream(), "num_unique_ids", parallel_id,
             num_unique_ids->shape().elem_cnt() * sizeof(IDX), num_unique_ids->dptr());
  DumpToFile(ctx->stream(), "ids_reverse_idx", parallel_id,
             ids_reverse_idx->shape().elem_cnt() * sizeof(IDX), ids_reverse_idx->dptr());
  DumpToFile(ctx->stream(), "cur_rank_num_unique_ids", parallel_id,
             cur_rank_num_unique_ids->shape().elem_cnt() * sizeof(IDX),
             cur_rank_num_unique_ids->dptr());
  DumpToFile(ctx->stream(), "cur_rank_unique_ids", parallel_id,
             cur_rank_unique_ids->shape().elem_cnt() * sizeof(K), cur_rank_unique_ids->dptr());
  DumpToFile(ctx->stream(), "cur_rank_reverse_idx", parallel_id,
             cur_rank_reverse_idx->shape().elem_cnt() * sizeof(IDX), cur_rank_reverse_idx->dptr());
  DumpToFile(ctx->stream(), "num_unique_ids_matrix", parallel_id,
             num_unique_ids_matrix->shape().elem_cnt() * sizeof(IDX),
             num_unique_ids_matrix->dptr());
}

struct BelongTo {
  int parallel_num;
  int parallel_id;
  __host__ __device__ __forceinline__ BelongTo(int parallel_num, int parallel_id)
      : parallel_num(parallel_num), parallel_id(parallel_id) {}
  __host__ __device__ __forceinline__ bool operator()(const int& a) const {
    return (a % parallel_num) == parallel_id;
  }
};

template<typename K, typename IDX>
void GetPartitionWorkspaceSizeInBytes(ep::Stream* stream, int64_t n, int64_t parallel_num,
                                      size_t* workspace_size_in_bytes) {
  BelongTo belong_to(parallel_num, 0);
  hipcub::DeviceSelect::If<K*, K*, IDX*>(nullptr, *workspace_size_in_bytes, nullptr, nullptr, nullptr,
                                      n, belong_to, stream->As<ep::CudaStream>()->hip_stream());
}

template<typename K, typename IDX>
void Partition(ep::Stream* stream, int64_t num_ids, IDX num_valid, int64_t parallel_num, K* in,
               K* out, IDX* num_out, void* workspace, size_t workspace_size_in_bytes) {
  for (int64_t i = 0; i < parallel_num; ++i) {
    BelongTo belong_to(parallel_num, i);
    hipcub::DeviceSelect::If(workspace, workspace_size_in_bytes, in, out + i * num_ids, num_out + i,
                          num_valid, belong_to, stream->As<ep::CudaStream>()->hip_stream());
  }
}

template<typename K, typename IDX>
class IdShuffleTmpBufferManager final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(IdShuffleTmpBufferManager);
  IdShuffleTmpBufferManager(void* ptr, const int64_t num_ids, const int64_t parallel_num)
      : ptr_(ptr) {
    int64_t unique_workspace_bytes = 0;
    UniqueKernelUtil<DeviceType::kCUDA, K, IDX>::GetUniqueWorkspaceSizeInBytes(
        nullptr, parallel_num * num_ids, &unique_workspace_bytes);
    // size_t partition_workspace_bytes = 0;
    // TODO: GetPartitionWorkspaceSizeInBytes have bug?
    // GetPartitionWorkspaceSizeInBytes<K, IDX>(nullptr, num_ids, parallel_num,
    //                                         &partition_workspace_bytes);
    // workspace_bytes_ = GetCudaAlignedSize(
    //    std::max(static_cast<size_t>(unique_workspace_bytes), partition_workspace_bytes));
    workspace_bytes_ = GetCudaAlignedSize(unique_workspace_bytes);
    const size_t unique_ids_bytes = GetCudaAlignedSize(num_ids * sizeof(K));
    const size_t partitioned_unique_ids_bytes =
        GetCudaAlignedSize(parallel_num * num_ids * sizeof(K));
    const size_t partitioned_num_unique_ids_bytes = GetCudaAlignedSize(parallel_num * sizeof(IDX));
    const size_t received_unique_ids_bytes = GetCudaAlignedSize(parallel_num * num_ids * sizeof(K));

    workspace_offset_ = 0;
    unique_ids_offset_ = workspace_offset_ + workspace_bytes_;
    partitioned_unique_ids_offset_ = unique_ids_offset_ + unique_ids_bytes;
    partitioned_num_unique_ids_offset_ =
        partitioned_unique_ids_offset_ + partitioned_unique_ids_bytes;
    received_unique_ids_offset_ =
        partitioned_num_unique_ids_offset_ + partitioned_num_unique_ids_bytes;
    CHECK_GE(workspace_bytes_, 0);
    total_buffer_size_ = workspace_bytes_ + unique_ids_bytes + partitioned_unique_ids_bytes
                         + partitioned_num_unique_ids_bytes + received_unique_ids_bytes;
  }
  ~IdShuffleTmpBufferManager() = default;

  int64_t WorkspaceBytes() const { return workspace_bytes_; }
  size_t TotalBufferSize() const { return total_buffer_size_; }

  void* WorkspacePtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<void*>(reinterpret_cast<char*>(ptr_) + workspace_offset_);
  }
  K* UniqueIdsPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + unique_ids_offset_);
  }
  K* PartitionedUniqueIdsPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + partitioned_unique_ids_offset_);
  }
  IDX* PartitionedNumUniqueIdsPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<IDX*>(reinterpret_cast<char*>(ptr_)
                                  + partitioned_num_unique_ids_offset_);
  }
  K* ReceivedUniqueIdsPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + received_unique_ids_offset_);
  }

 private:
  size_t workspace_offset_;
  size_t unique_ids_offset_;
  size_t partitioned_unique_ids_offset_;
  size_t partitioned_num_unique_ids_offset_;
  size_t received_unique_ids_offset_;

  size_t workspace_bytes_;
  size_t total_buffer_size_;
  void* ptr_;
};

class IdShuffleNcclKernelCommState final : public user_op::OpKernelState {
 public:
  explicit IdShuffleNcclKernelCommState(user_op::KernelInitContext* ctx)
      : is_init_(false), parallel_desc_(ctx->parallel_desc()) {}
  ~IdShuffleNcclKernelCommState() = default;

  ncclComm_t comm() {
    if (!is_init_) {
      std::set<std::pair<int64_t, int64_t>> device_set;
      FOR_RANGE(int64_t, parallel_id, 0, parallel_desc_.parallel_num()) {
        int64_t machine_id = CHECK_JUST(parallel_desc_.MachineId4ParallelId(parallel_id));
        int64_t device_id = CHECK_JUST(parallel_desc_.DeviceId4ParallelId(parallel_id));
        device_set.emplace(std::make_pair(machine_id, device_id));
      }
      EagerNcclCommMgr* comm_mgr = CHECK_NOTNULL(Global<EagerNcclCommMgr>::Get());
      comm_ = comm_mgr->GetCommForDeviceAndStreamName(device_set, "ID_SHUFFLE");
      is_init_ = true;
    }
    return comm_;
  }

 private:
  bool is_init_;
  ParallelDesc parallel_desc_;
  ncclComm_t comm_{};
};

class NcclKernelCommState final : public user_op::OpKernelState {
 public:
  explicit NcclKernelCommState(user_op::KernelInitContext* ctx)
      : is_init_(false), parallel_desc_(ctx->parallel_desc()) {}
  ~NcclKernelCommState() = default;

  ncclComm_t comm() {
    if (!is_init_) {
      std::set<std::pair<int64_t, int64_t>> device_set;
      FOR_RANGE(int64_t, parallel_id, 0, parallel_desc_.parallel_num()) {
        int64_t machine_id = CHECK_JUST(parallel_desc_.MachineId4ParallelId(parallel_id));
        int64_t device_id = CHECK_JUST(parallel_desc_.DeviceId4ParallelId(parallel_id));
        device_set.emplace(std::make_pair(machine_id, device_id));
      }
      EagerNcclCommMgr* comm_mgr = CHECK_NOTNULL(Global<EagerNcclCommMgr>::Get());
      comm_ = comm_mgr->GetCommForDevice(device_set);
      is_init_ = true;
    }
    return comm_;
  }

 private:
  bool is_init_;
  ParallelDesc parallel_desc_;
  ncclComm_t comm_{};
};

}  // namespace

template<typename K, typename IDX>
class IdShuffleKernel final : public user_op::OpKernel {
 public:
  IdShuffleKernel() = default;
  ~IdShuffleKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<IdShuffleNcclKernelCommState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* nccl_comm = dynamic_cast<IdShuffleNcclKernelCommState*>(state);
    CHECK(nccl_comm != nullptr);
    const user_op::Tensor* ids = ctx->Tensor4ArgNameAndIndex("ids", 0);
    user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    user_op::Tensor* ids_reverse_idx = ctx->Tensor4ArgNameAndIndex("ids_reverse_idx", 0);
    user_op::Tensor* cur_rank_num_unique_ids =
        ctx->Tensor4ArgNameAndIndex("cur_rank_num_unique_ids", 0);
    user_op::Tensor* cur_rank_unique_ids = ctx->Tensor4ArgNameAndIndex("cur_rank_unique_ids", 0);
    user_op::Tensor* cur_rank_reverse_idx = ctx->Tensor4ArgNameAndIndex("cur_rank_reverse_idx", 0);
    user_op::Tensor* num_unique_ids_matrix =
        ctx->Tensor4ArgNameAndIndex("num_unique_ids_matrix", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t num_ids = ids->shape().elem_cnt();
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    hipStream_t hip_stream = ctx->stream()->As<ep::CudaStream>()->hip_stream();
    IDX* host_num_unique_ids;
    OF_CUDA_CHECK(
        hipHostMalloc(&host_num_unique_ids, (parallel_num * parallel_num + 1) * sizeof(IDX)));

    IdShuffleTmpBufferManager<K, IDX> buffer_manager(tmp_buffer->mut_dptr(), num_ids, parallel_num);
    void* workspace_ptr = buffer_manager.WorkspacePtr();
    size_t workspace_size = buffer_manager.WorkspaceBytes();
    // unique
    UniqueKernelUtil<DeviceType::kCUDA, K, IDX>::Unique(
        ctx->stream(), num_ids, ids->dptr<K>(), num_unique_ids->mut_dptr<IDX>(),
        buffer_manager.UniqueIdsPtr(), ids_reverse_idx->mut_dptr<IDX>(), workspace_ptr,
        workspace_size);
    // partition
    std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
        ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                  ep::primitive::MemcpyKind::kDtoH);
    CHECK(copyd2h_primitive);
    copyd2h_primitive->Launch(ctx->stream(), host_num_unique_ids, num_unique_ids->mut_dptr(),
                              sizeof(IDX));
    CHECK_JUST(ctx->stream()->Sync());

    LOG(ERROR) << "rank " << parallel_id << " num_unique_ids " << *host_num_unique_ids;
    K* partitioned_unique_ids = buffer_manager.PartitionedUniqueIdsPtr();
    IDX* partitioned_num_unique_ids = buffer_manager.PartitionedNumUniqueIdsPtr();
    K* received_unique_ids = buffer_manager.ReceivedUniqueIdsPtr();
    /*
    num_unique_ids_matrix(parallel_num * parallel_num):
           partion0   partion1
    rank0
    rank1
    */
    IDX* received_num_unique_ids_matrix = num_unique_ids_matrix->mut_dptr<IDX>();
    Partition(ctx->stream(), num_ids, host_num_unique_ids[0], parallel_num,
              buffer_manager.UniqueIdsPtr(), partitioned_unique_ids, partitioned_num_unique_ids,
              workspace_ptr, workspace_size);

    // allgather count
    ncclComm_t comm = nccl_comm->comm();
    OF_NCCL_CHECK(ncclAllGather(reinterpret_cast<const void*>(partitioned_num_unique_ids),
                                reinterpret_cast<void*>(received_num_unique_ids_matrix),
                                parallel_num, GetNcclDataType(cur_rank_num_unique_ids->data_type()),
                                comm, hip_stream));
    IDX* host_num_unique_ids_matrix = host_num_unique_ids + 1;
    copyd2h_primitive->Launch(ctx->stream(), host_num_unique_ids_matrix,
                              received_num_unique_ids_matrix,
                              parallel_num * parallel_num * sizeof(IDX));
    CHECK_JUST(ctx->stream()->Sync());

    // send recv
    int64_t recv_offset = 0;
    OF_NCCL_CHECK(ncclGroupStart());
    for (int64_t j = 0; j < parallel_num; ++j) {
      const int64_t need_send_elem_cnt = host_num_unique_ids_matrix[parallel_id * parallel_num + j];
      const int64_t need_recv_elem_cnt = host_num_unique_ids_matrix[j * parallel_num + parallel_id];
      OF_NCCL_CHECK(ncclSend(reinterpret_cast<const void*>(partitioned_unique_ids + j * num_ids),
                             need_send_elem_cnt, GetNcclDataType(ids->data_type()), j, comm,
                             hip_stream));
      OF_NCCL_CHECK(ncclRecv(reinterpret_cast<void*>(received_unique_ids + recv_offset),
                             need_recv_elem_cnt, GetNcclDataType(ids->data_type()), j, comm,
                             hip_stream));
      recv_offset += need_recv_elem_cnt;
    }
    OF_NCCL_CHECK(ncclGroupEnd());
    // unique
    UniqueKernelUtil<DeviceType::kCUDA, K, IDX>::Unique(
        ctx->stream(), recv_offset, received_unique_ids, cur_rank_num_unique_ids->mut_dptr<IDX>(),
        cur_rank_unique_ids->mut_dptr<K>(), cur_rank_reverse_idx->mut_dptr<IDX>(), workspace_ptr,
        workspace_size);

    if (ParseBooleanFromEnv("DEBUG_SHUFFLE", false)) { DebugIdShuffle<K, IDX>(ctx); }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename K, typename IDX>
user_op::InferTmpSizeFn GenIdShuffleInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    const user_op::TensorDesc& ids = ctx->InputTensorDesc("ids", 0);
    IdShuffleTmpBufferManager<K, IDX> buffer_manager(nullptr, ids.shape().elem_cnt(),
                                                     ctx->parallel_desc().parallel_num());
    return buffer_manager.TotalBufferSize();
  };
}

#define REGISTER_CUDA_ID_SHUFFLE_KERNEL(k_dtype, idx_dtype)                                \
  REGISTER_USER_KERNEL("id_shuffle")                                                       \
      .SetCreateFn<IdShuffleKernel<k_dtype, idx_dtype>>()                                  \
      .SetIsMatchedHob(                                                                    \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                  \
          && (user_op::HobDataType("ids", 0) == GetDataType<k_dtype>::value)               \
          && (user_op::HobDataType("num_unique_ids", 0) == GetDataType<idx_dtype>::value)) \
      .SetInferTmpSizeFn(GenIdShuffleInferTmpSizeFn<k_dtype, idx_dtype>());

// REGISTER_CUDA_ID_SHUFFLE_KERNEL(int32_t, int32_t)
REGISTER_CUDA_ID_SHUFFLE_KERNEL(int64_t, int32_t)

template<typename T, typename IDX>
class EmbeddingShuffleKernel final : public user_op::OpKernel {
 public:
  EmbeddingShuffleKernel() = default;
  ~EmbeddingShuffleKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<NcclKernelCommState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* nccl_comm = dynamic_cast<NcclKernelCommState*>(state);
    CHECK(nccl_comm != nullptr);
    ncclComm_t comm = nccl_comm->comm();

    LOG(ERROR) << "EmbeddingShuffleKernel";
    const user_op::Tensor* cur_rank_embeddings =
        ctx->Tensor4ArgNameAndIndex("cur_rank_embeddings", 0);
    const user_op::Tensor* cur_rank_num_unique_ids =
        ctx->Tensor4ArgNameAndIndex("cur_rank_num_unique_ids", 0);
    const user_op::Tensor* cur_rank_reverse_idx =
        ctx->Tensor4ArgNameAndIndex("cur_rank_reverse_idx", 0);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* ids_reverse_idx = ctx->Tensor4ArgNameAndIndex("ids_reverse_idx", 0);
    const user_op::Tensor* num_unique_ids_matrix =
        ctx->Tensor4ArgNameAndIndex("num_unique_ids_matrix", 0);
    user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t num_ids = ids_reverse_idx->shape().elem_cnt();
    const int64_t embedding_size = embeddings->shape().elem_cnt() / num_ids;
    hipStream_t hip_stream = ctx->stream()->As<ep::CudaStream>()->hip_stream();

    IDX* host_num_unique_ids_matrix;
    OF_CUDA_CHECK(
        hipHostMalloc(&host_num_unique_ids_matrix, parallel_num * parallel_num * sizeof(IDX)));
    std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
        ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                  ep::primitive::MemcpyKind::kDtoH);
    CHECK(copyd2h_primitive);
    copyd2h_primitive->Launch(ctx->stream(), host_num_unique_ids_matrix,
                              num_unique_ids_matrix->dptr(),
                              parallel_num * parallel_num * sizeof(IDX));
    CHECK_JUST(ctx->stream()->Sync());
    int64_t cur_rank_num_ids = 0;
    for (int64_t i = 0; i < parallel_num; ++i) {
      cur_rank_num_ids += host_num_unique_ids_matrix[i * parallel_num + parallel_id];
    }

    LOG(ERROR) << "parallel_id " << parallel_id << " cur_rank_num_ids " << cur_rank_num_ids;
    size_t reverse_cur_rank_embeddings_size =
        GetCudaAlignedSize(cur_rank_num_ids * embedding_size * sizeof(T));
    T* reverse_cur_rank_embeddings = reinterpret_cast<T*>(tmp_buffer->mut_dptr());
    T* recv_unique_embeddings =
        reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + reverse_cur_rank_embeddings_size);

    GatherKernelUtilImpl<DeviceType::kCUDA, T, IDX>::Forward(
        ctx->stream(), cur_rank_reverse_idx->dptr<IDX>(), cur_rank_num_ids,
        cur_rank_embeddings->dptr<T>(),
        Shape({1, cur_rank_embeddings->shape().elem_cnt() / embedding_size, embedding_size}),
        reverse_cur_rank_embeddings, 0);
    int64_t send_offset = 0;
    int64_t recv_offset = 0;
    OF_NCCL_CHECK(ncclGroupStart());
    for (int64_t i = 0; i < parallel_num; ++i) {
      const int64_t need_send_elem_cnt =
          host_num_unique_ids_matrix[i * parallel_num + parallel_id] * embedding_size;
      const int64_t need_recv_elem_cnt =
          host_num_unique_ids_matrix[parallel_id * parallel_num + i] * embedding_size;
      LOG(ERROR) << "need_send_elem_cnt "
                 << host_num_unique_ids_matrix[i * parallel_num + parallel_id];
      LOG(ERROR) << " need_recv_elem_cnt "
                 << host_num_unique_ids_matrix[parallel_id * parallel_num + i];
      OF_NCCL_CHECK(
          ncclSend(reinterpret_cast<const void*>(reverse_cur_rank_embeddings + send_offset),
                   need_send_elem_cnt, GetNcclDataType(cur_rank_embeddings->data_type()), i, comm,
                   hip_stream));
      OF_NCCL_CHECK(ncclRecv(reinterpret_cast<void*>(recv_unique_embeddings + recv_offset),
                             need_recv_elem_cnt, GetNcclDataType(cur_rank_embeddings->data_type()),
                             i, comm, hip_stream));
      send_offset += need_send_elem_cnt;
      recv_offset += need_recv_elem_cnt;
    }
    OF_NCCL_CHECK(ncclGroupEnd());
    GatherKernelUtilImpl<DeviceType::kCUDA, T, IDX>::Forward(
        ctx->stream(), ids_reverse_idx->dptr<IDX>(), ids_reverse_idx->shape().elem_cnt(),
        recv_unique_embeddings, Shape({1, num_ids, embedding_size}), embeddings->mut_dptr<T>(), 0);

    if (ParseBooleanFromEnv("DEBUG_SHUFFLE", false)) { DebugEmbeddingShuffle<T>(ctx); }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T, typename IDX>
user_op::InferTmpSizeFn GenEmbeddingShuffleInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    const user_op::TensorDesc& cur_rank_embeddings = ctx->InputTensorDesc("cur_rank_embeddings", 0);
    const user_op::TensorDesc& embeddings = ctx->InputTensorDesc("embeddings", 0);
    size_t reverse_cur_rank_embeddings_size =
        GetCudaAlignedSize(cur_rank_embeddings.shape().elem_cnt() * sizeof(T));
    size_t recv_unique_embeddings = GetCudaAlignedSize(embeddings.shape().elem_cnt() * sizeof(T));
    return reverse_cur_rank_embeddings_size + recv_unique_embeddings;
  };
}

#define REGISTER_CUDA_EMBEDDING_SHUFFLE_KERNEL(t_dtype, idx_dtype)                                 \
  REGISTER_USER_KERNEL("embedding_shuffle")                                                        \
      .SetCreateFn<EmbeddingShuffleKernel<t_dtype, idx_dtype>>()                                   \
      .SetIsMatchedHob(                                                                            \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                          \
          && (user_op::HobDataType("cur_rank_num_unique_ids", 0) == GetDataType<idx_dtype>::value) \
          && (user_op::HobDataType("embeddings", 0) == GetDataType<t_dtype>::value))               \
      .SetInferTmpSizeFn(GenEmbeddingShuffleInferTmpSizeFn<t_dtype, idx_dtype>());

REGISTER_CUDA_EMBEDDING_SHUFFLE_KERNEL(float, int32_t)

template<typename T, typename IDX>
class EmbeddingGradientShuffleKernel final : public user_op::OpKernel {
 public:
  EmbeddingGradientShuffleKernel() = default;
  ~EmbeddingGradientShuffleKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<NcclKernelCommState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* nccl_comm = dynamic_cast<NcclKernelCommState*>(state);
    CHECK(nccl_comm != nullptr);
    ncclComm_t comm = nccl_comm->comm();
    LOG(ERROR) << "EmbeddingGradientShuffleKernel";

    const user_op::Tensor* embedding_diff = ctx->Tensor4ArgNameAndIndex("embedding_diff", 0);
    const user_op::Tensor* cur_rank_num_unique_ids =
        ctx->Tensor4ArgNameAndIndex("cur_rank_num_unique_ids", 0);
    const user_op::Tensor* cur_rank_reverse_idx =
        ctx->Tensor4ArgNameAndIndex("cur_rank_reverse_idx", 0);
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* ids_reverse_idx = ctx->Tensor4ArgNameAndIndex("ids_reverse_idx", 0);
    const user_op::Tensor* num_unique_ids_matrix =
        ctx->Tensor4ArgNameAndIndex("num_unique_ids_matrix", 0);
    user_op::Tensor* cur_rank_unique_embedding_diff =
        ctx->Tensor4ArgNameAndIndex("cur_rank_unique_embedding_diff", 0);
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t num_ids = ids_reverse_idx->shape().elem_cnt();
    const int64_t embedding_size = embedding_diff->shape().elem_cnt() / num_ids;
    hipStream_t hip_stream = ctx->stream()->As<ep::CudaStream>()->hip_stream();
    IDX* host_num_unique_ids_matrix;
    OF_CUDA_CHECK(
        hipHostMalloc(&host_num_unique_ids_matrix, parallel_num * parallel_num * sizeof(IDX)));
    std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
        ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                  ep::primitive::MemcpyKind::kDtoH);
    CHECK(copyd2h_primitive);
    copyd2h_primitive->Launch(ctx->stream(), host_num_unique_ids_matrix,
                              num_unique_ids_matrix->dptr(),
                              parallel_num * parallel_num * sizeof(IDX));
    CHECK_JUST(ctx->stream()->Sync());
    int64_t cur_rank_num_ids = 0;
    for (int64_t i = 0; i < parallel_num; ++i) {
      cur_rank_num_ids += host_num_unique_ids_matrix[i * parallel_num + parallel_id];
    }

    size_t unique_diff_size = embedding_diff->shape().elem_cnt() * sizeof(T);
    T* unique_diff_ptr = reinterpret_cast<T*>(tmp_buffer->mut_dptr());
    T* recv_embeddings_diff = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + unique_diff_size);

    Memset<DeviceType::kCUDA>(ctx->stream(), unique_diff_ptr, 0,
                              embedding_diff->shape().elem_cnt() * sizeof(T));
    UnsortedSegmentSumKernelUtil<DeviceType::kCUDA, T, IDX, T>::UnsortedSegmentSum(
        ctx->stream(), ids_reverse_idx->dptr<IDX>(), embedding_diff->dptr<T>(), num_ids, num_ids, 1,
        embedding_size, 0, unique_diff_ptr);

    int64_t send_offset = 0;
    int64_t recv_offset = 0;
    OF_NCCL_CHECK(ncclGroupStart());
    for (int64_t i = 0; i < parallel_num; ++i) {
      const int64_t need_send_elem_cnt =
          host_num_unique_ids_matrix[parallel_id * parallel_num + i] * embedding_size;
      const int64_t need_recv_elem_cnt =
          host_num_unique_ids_matrix[i * parallel_num + parallel_id] * embedding_size;
      OF_NCCL_CHECK(ncclSend(reinterpret_cast<const void*>(unique_diff_ptr + send_offset),
                             need_send_elem_cnt, GetNcclDataType(embedding_diff->data_type()), i,
                             comm, hip_stream));
      OF_NCCL_CHECK(ncclRecv(reinterpret_cast<void*>(recv_embeddings_diff + recv_offset),
                             need_recv_elem_cnt, GetNcclDataType(embedding_diff->data_type()), i,
                             comm, hip_stream));
      send_offset += need_send_elem_cnt;
      recv_offset += need_recv_elem_cnt;
    }
    OF_NCCL_CHECK(ncclGroupEnd());

    Memset<DeviceType::kCUDA>(ctx->stream(), cur_rank_unique_embedding_diff->mut_dptr<T>(), 0,
                              cur_rank_unique_embedding_diff->shape().elem_cnt() * sizeof(T));
    UnsortedSegmentSumKernelUtil<DeviceType::kCUDA, T, IDX, T>::UnsortedSegmentSum(
        ctx->stream(), cur_rank_reverse_idx->dptr<IDX>(), recv_embeddings_diff, cur_rank_num_ids,
        cur_rank_num_ids, 1, embedding_size, 0, cur_rank_unique_embedding_diff->mut_dptr<T>());

    if (ParseBooleanFromEnv("DEBUG_SHUFFLE", false)) { DebugEmbeddingGradientShuffle<T>(ctx); }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T, typename IDX>
user_op::InferTmpSizeFn GenEmbeddingGradientShuffleInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    const user_op::TensorDesc& embedding_diff = ctx->InputTensorDesc("embedding_diff", 0);
    const user_op::TensorDesc& cur_rank_unique_embedding_diff =
        ctx->InputTensorDesc("cur_rank_unique_embedding_diff", 0);
    size_t unique_embedding_diff_size = embedding_diff.shape().elem_cnt() * sizeof(T);
    size_t cur_rank_unique_embedding_diff_size =
        cur_rank_unique_embedding_diff.shape().elem_cnt() * sizeof(T);
    return unique_embedding_diff_size + cur_rank_unique_embedding_diff_size;
  };
}

#define REGISTER_CUDA_EMBEDDING_GRADIENT_SHUFFLE_KERNEL(t_dtype, idx_dtype)                \
  REGISTER_USER_KERNEL("embedding_gradient_shuffle")                                       \
      .SetCreateFn<EmbeddingGradientShuffleKernel<t_dtype, idx_dtype>>()                   \
      .SetIsMatchedHob(                                                                    \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                  \
          && (user_op::HobDataType("ids_reverse_idx", 0) == GetDataType<idx_dtype>::value) \
          && (user_op::HobDataType("embedding_diff", 0) == GetDataType<t_dtype>::value))   \
      .SetInferTmpSizeFn(GenEmbeddingGradientShuffleInferTmpSizeFn<t_dtype, idx_dtype>());

REGISTER_CUDA_EMBEDDING_GRADIENT_SHUFFLE_KERNEL(float, int32_t)

}  // namespace oneflow
