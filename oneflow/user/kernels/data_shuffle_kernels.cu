#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/unique_kernel_util.h"
#include "oneflow/core/cuda/unique.cuh"
#include "oneflow/core/device/nccl_util.h"
#include "oneflow/core/job/eager_nccl_comm_manager.h"
#include "oneflow/core/job/parallel_desc.h"
#include "oneflow/core/ep/include/primitive/memcpy.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/user/kernels/gather_kernel_util.h"
#include "oneflow/user/kernels/unsorted_segment_sum_kernel_util.h"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/core/ep/include/primitive/cast.h"
#include "oneflow/user/kernels/unique_key.cuh"

namespace oneflow {

namespace {

void DumpToFile(ep::Stream* stream, std::string filename, int64_t parallel_id, size_t data_size,
                const void* ptr) {
  void* host_ptr;
  OF_CUDA_CHECK(hipHostMalloc(&host_ptr, data_size));
  std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
      ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                ep::primitive::MemcpyKind::kDtoH);
  CHECK(copyd2h_primitive);
  copyd2h_primitive->Launch(stream, host_ptr, ptr, data_size);
  CHECK_JUST(stream->Sync());
  std::ofstream dx_os;
  dx_os.open(StrCat("test/" + filename + "_", parallel_id));
  dx_os.write(reinterpret_cast<char*>(host_ptr), data_size);
  dx_os.close();
  OF_CUDA_CHECK(hipHostFree(host_ptr));
}

template<typename T>
void DebugEmbeddingShuffle(user_op::KernelComputeContext* ctx) {
  const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
  user_op::Tensor* cur_rank_embeddings = ctx->Tensor4ArgNameAndIndex("cur_rank_embeddings", 0);
  user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
  DumpToFile(ctx->stream(), "cur_rank_embeddings", parallel_id,
             cur_rank_embeddings->shape().elem_cnt() * sizeof(T), cur_rank_embeddings->dptr());
  DumpToFile(ctx->stream(), "embeddings", parallel_id, embeddings->shape().elem_cnt() * sizeof(T),
             embeddings->dptr());
}

template<typename T>
void DebugEmbeddingGradientShuffle(user_op::KernelComputeContext* ctx) {
  const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
  user_op::Tensor* cur_rank_unique_embedding_diff =
      ctx->Tensor4ArgNameAndIndex("cur_rank_unique_embedding_diff", 0);
  user_op::Tensor* embedding_diff = ctx->Tensor4ArgNameAndIndex("embedding_diff", 0);
  DumpToFile(ctx->stream(), "cur_rank_unique_embedding_diff", parallel_id,
             cur_rank_unique_embedding_diff->shape().elem_cnt() * sizeof(T),
             cur_rank_unique_embedding_diff->dptr());
  DumpToFile(ctx->stream(), "embedding_diff", parallel_id,
             embedding_diff->shape().elem_cnt() * sizeof(T), embedding_diff->dptr());
}

template<typename K, typename IDX>
void DebugIdShuffle(user_op::KernelComputeContext* ctx) {
  const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
  const user_op::Tensor* ids = ctx->Tensor4ArgNameAndIndex("ids", 0);
  user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
  user_op::Tensor* ids_reverse_idx = ctx->Tensor4ArgNameAndIndex("ids_reverse_idx", 0);
  user_op::Tensor* cur_rank_num_unique_ids =
      ctx->Tensor4ArgNameAndIndex("cur_rank_num_unique_ids", 0);
  user_op::Tensor* cur_rank_unique_ids = ctx->Tensor4ArgNameAndIndex("cur_rank_unique_ids", 0);
  user_op::Tensor* cur_rank_reverse_idx = ctx->Tensor4ArgNameAndIndex("cur_rank_reverse_idx", 0);
  user_op::Tensor* num_unique_ids_matrix = ctx->Tensor4ArgNameAndIndex("num_unique_ids_matrix", 0);
  DumpToFile(ctx->stream(), "ids", parallel_id, ids->shape().elem_cnt() * sizeof(K), ids->dptr());
  DumpToFile(ctx->stream(), "num_unique_ids", parallel_id,
             num_unique_ids->shape().elem_cnt() * sizeof(IDX), num_unique_ids->dptr());
  DumpToFile(ctx->stream(), "ids_reverse_idx", parallel_id,
             ids_reverse_idx->shape().elem_cnt() * sizeof(IDX), ids_reverse_idx->dptr());
  DumpToFile(ctx->stream(), "cur_rank_num_unique_ids", parallel_id,
             cur_rank_num_unique_ids->shape().elem_cnt() * sizeof(IDX),
             cur_rank_num_unique_ids->dptr());
  DumpToFile(ctx->stream(), "cur_rank_unique_ids", parallel_id,
             cur_rank_unique_ids->shape().elem_cnt() * sizeof(K), cur_rank_unique_ids->dptr());
  DumpToFile(ctx->stream(), "cur_rank_reverse_idx", parallel_id,
             cur_rank_reverse_idx->shape().elem_cnt() * sizeof(IDX), cur_rank_reverse_idx->dptr());
  DumpToFile(ctx->stream(), "num_unique_ids_matrix", parallel_id,
             num_unique_ids_matrix->shape().elem_cnt() * sizeof(IDX),
             num_unique_ids_matrix->dptr());
  user_op::Tensor* partition_index = ctx->Tensor4ArgNameAndIndex("partition_index", 0);
  DumpToFile(ctx->stream(), "partition_index", parallel_id,
             partition_index->shape().elem_cnt() * sizeof(IDX), partition_index->dptr());

  user_op::Tensor* cur_rank_column_ids = ctx->Tensor4ArgNameAndIndex("cur_rank_column_ids", 0);
  DumpToFile(ctx->stream(), "cur_rank_column_ids", parallel_id,
             cur_rank_column_ids->shape().elem_cnt() * sizeof(IDX), cur_rank_column_ids->dptr());
}

template<typename K, typename IDX>
__global__ void PartitionKernel(int64_t n, const IDX num_ids, const int parallel_num, const K* ids,
                                const IDX* column_ids, K* out_ids, IDX* out_column_ids,
                                IDX* num_out, IDX* out_index) {
  CUDA_1D_KERNEL_LOOP(i, num_ids) {
    const K id = ids[i];
    int64_t partition_id = id % parallel_num;
    IDX old_key_offset = cuda::atomic::Add(num_out + partition_id, 1);
    IDX offset = partition_id * n + old_key_offset;
    out_ids[offset] = id;
    out_index[offset] = i;
    out_column_ids[offset] = column_ids[i];
  }
}

template<typename T, typename IDX>
__global__ void ReversePartitionEmbeddingKernel(int64_t n, int64_t embedding_size,
                                                const IDX* partition_index, const T* recv_embedding,
                                                T* reverse_partition_embeddings) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    const IDX row = i / embedding_size;
    const IDX col = i - row * embedding_size;
    const IDX partition_id = partition_index[row];
    reverse_partition_embeddings[partition_id * embedding_size + col] = recv_embedding[i];
  }
}

template<typename T, typename IDX>
__global__ void PartitionEmbeddingKernel(int64_t n, int64_t embedding_size,
                                         const IDX* partition_index, const T* embeddings,
                                         T* partition_embeddings) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    const IDX row = i / embedding_size;
    const IDX col = i - row * embedding_size;
    const IDX partition_id = partition_index[row];
    partition_embeddings[i] = embeddings[partition_id * embedding_size + col];
  }
}

template<typename K, typename IDX>
void Partition(ep::Stream* stream, int64_t num_ids, IDX num_valid, int64_t parallel_num, K* in,
               IDX* column_ids, K* out, IDX* out_column_ids, IDX* num_out, IDX* out_index) {
  OF_CUDA_CHECK(hipMemset(num_out, 0, parallel_num * sizeof(IDX)));
  PartitionKernel<<<BlocksNum4ThreadsNum(num_ids), kCudaThreadsNumPerBlock, 0,
                    stream->As<ep::CudaStream>()->hip_stream()>>>(
      num_ids, num_valid, parallel_num, in, column_ids, out, out_column_ids, num_out, out_index);
}

template<typename K, typename IDX>
class IdShuffleTmpBufferManager final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(IdShuffleTmpBufferManager);
  IdShuffleTmpBufferManager(void* ptr, const int64_t num_ids, const int64_t parallel_num)
      : ptr_(ptr) {
    int64_t unique_workspace_bytes = 0;
    workspace_bytes_ =
        GetUniqueKeysWorkspace<K, IDX>(parallel_num * num_ids, parallel_num * num_ids);
    const size_t unique_ids_bytes = GetCudaAlignedSize(num_ids * sizeof(K));
    const size_t partitioned_unique_ids_bytes =
        GetCudaAlignedSize(parallel_num * num_ids * sizeof(K));
    const size_t partitioned_num_unique_ids_bytes = GetCudaAlignedSize(parallel_num * sizeof(IDX));
    const size_t received_unique_ids_bytes = GetCudaAlignedSize(parallel_num * num_ids * sizeof(K));
    const size_t unique_column_ids_bytes = GetCudaAlignedSize(num_ids * sizeof(IDX));
    const size_t partitioned_column_ids_bytes =
        GetCudaAlignedSize(parallel_num * num_ids * sizeof(IDX));
    const size_t received_column_ids_bytes =
        GetCudaAlignedSize(parallel_num * num_ids * sizeof(IDX));

    workspace_offset_ = 0;
    unique_ids_offset_ = workspace_offset_ + workspace_bytes_;
    partitioned_unique_ids_offset_ = unique_ids_offset_ + unique_ids_bytes;
    partitioned_num_unique_ids_offset_ =
        partitioned_unique_ids_offset_ + partitioned_unique_ids_bytes;
    received_unique_ids_offset_ =
        partitioned_num_unique_ids_offset_ + partitioned_num_unique_ids_bytes;
    unique_column_ids_offset_ = received_unique_ids_offset_ + received_unique_ids_bytes;
    partitioned_column_ids_offset_ = unique_column_ids_offset_ + unique_column_ids_bytes;
    received_column_ids_offset_ = partitioned_column_ids_offset_ + partitioned_column_ids_bytes;

    CHECK_GE(workspace_bytes_, 0);
    total_buffer_size_ = workspace_bytes_ + unique_ids_bytes + partitioned_unique_ids_bytes
                         + partitioned_num_unique_ids_bytes + received_unique_ids_bytes
                         + unique_column_ids_bytes + partitioned_column_ids_bytes
                         + received_column_ids_bytes;
  }
  ~IdShuffleTmpBufferManager() = default;

  int64_t WorkspaceBytes() const { return workspace_bytes_; }
  size_t TotalBufferSize() const { return total_buffer_size_; }

  void* WorkspacePtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<void*>(reinterpret_cast<char*>(ptr_) + workspace_offset_);
  }
  K* UniqueIdsPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + unique_ids_offset_);
  }
  IDX* UniqueColumnIdsPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<IDX*>(reinterpret_cast<char*>(ptr_) + unique_column_ids_offset_);
  }
  K* PartitionedUniqueIdsPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + partitioned_unique_ids_offset_);
  }
  IDX* PartitionedColumnIdsPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<IDX*>(reinterpret_cast<char*>(ptr_) + partitioned_column_ids_offset_);
  }
  IDX* PartitionedNumUniqueIdsPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<IDX*>(reinterpret_cast<char*>(ptr_)
                                  + partitioned_num_unique_ids_offset_);
  }
  K* ReceivedUniqueIdsPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + received_unique_ids_offset_);
  }
  IDX* ReceivedColumnIdsPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<IDX*>(reinterpret_cast<char*>(ptr_) + received_column_ids_offset_);
  }

 private:
  size_t workspace_offset_;
  size_t unique_ids_offset_;
  size_t partitioned_unique_ids_offset_;
  size_t partitioned_num_unique_ids_offset_;
  size_t received_unique_ids_offset_;
  size_t unique_column_ids_offset_;
  size_t partitioned_column_ids_offset_;
  size_t received_column_ids_offset_;

  size_t workspace_bytes_;
  size_t total_buffer_size_;
  void* ptr_;
};

class NcclKernelCommState final : public user_op::OpKernelState {
 public:
  explicit NcclKernelCommState(user_op::KernelInitContext* ctx)
      : is_init_(false),
        has_independent_stream_(ctx->op_conf().has_stream_name_hint()),
        stream_name_(""),
        parallel_desc_(ctx->parallel_desc()) {
    if (has_independent_stream_) { stream_name_ = ctx->op_conf().stream_name_hint(); }
    OF_CUDA_CHECK(hipHostMalloc(&host_num_unique_ids_matrix_,
                                 parallel_desc_.parallel_num() * parallel_desc_.parallel_num()
                                     * sizeof(int32_t)));  // TODO: int32_t->IDX
  }
  ~NcclKernelCommState() { OF_CUDA_CHECK(hipHostFree(host_num_unique_ids_matrix_)); }

  ncclComm_t comm() {
    if (!is_init_) {
      std::set<std::pair<int64_t, int64_t>> device_set;
      FOR_RANGE(int64_t, parallel_id, 0, parallel_desc_.parallel_num()) {
        int64_t machine_id = CHECK_JUST(parallel_desc_.MachineId4ParallelId(parallel_id));
        int64_t device_id = CHECK_JUST(parallel_desc_.DeviceId4ParallelId(parallel_id));
        device_set.emplace(std::make_pair(machine_id, device_id));
      }
      EagerNcclCommMgr* comm_mgr = CHECK_NOTNULL(Global<EagerNcclCommMgr>::Get());
      if (has_independent_stream_) {
        comm_ = comm_mgr->GetCommForDeviceAndStreamName(device_set, stream_name_);
      } else {
        comm_ = comm_mgr->GetCommForDevice(device_set);
      }
      is_init_ = true;
    }
    return comm_;
  }
  void* HostNumUniqueIdsMatrix() { return host_num_unique_ids_matrix_; }

 private:
  bool is_init_;
  bool has_independent_stream_;
  std::string stream_name_;
  ParallelDesc parallel_desc_;
  ncclComm_t comm_{};
  void* host_num_unique_ids_matrix_;
};

}  // namespace

template<typename K, typename IDX>
class IdShuffleKernel final : public user_op::OpKernel {
 public:
  IdShuffleKernel() = default;
  ~IdShuffleKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<NcclKernelCommState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* nccl_comm = dynamic_cast<NcclKernelCommState*>(state);
    CHECK(nccl_comm != nullptr);
    const user_op::Tensor* ids = ctx->Tensor4ArgNameAndIndex("ids", 0);
    const user_op::Tensor* column_ids = ctx->Tensor4ArgNameAndIndex("column_ids", 0);
    user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    user_op::Tensor* ids_reverse_idx = ctx->Tensor4ArgNameAndIndex("ids_reverse_idx", 0);
    user_op::Tensor* cur_rank_num_unique_ids =
        ctx->Tensor4ArgNameAndIndex("cur_rank_num_unique_ids", 0);
    user_op::Tensor* cur_rank_unique_ids = ctx->Tensor4ArgNameAndIndex("cur_rank_unique_ids", 0);
    user_op::Tensor* cur_rank_column_ids = ctx->Tensor4ArgNameAndIndex("cur_rank_column_ids", 0);
    user_op::Tensor* cur_rank_reverse_idx = ctx->Tensor4ArgNameAndIndex("cur_rank_reverse_idx", 0);
    user_op::Tensor* num_unique_ids_matrix =
        ctx->Tensor4ArgNameAndIndex("num_unique_ids_matrix", 0);
    user_op::Tensor* partition_index = ctx->Tensor4ArgNameAndIndex("partition_index", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t num_ids = ids->shape().elem_cnt();
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    hipStream_t hip_stream = ctx->stream()->As<ep::CudaStream>()->hip_stream();
    IDX* host_num_unique_ids = reinterpret_cast<IDX*>(nccl_comm->HostNumUniqueIdsMatrix());
    IdShuffleTmpBufferManager<K, IDX> buffer_manager(tmp_buffer->mut_dptr(), num_ids, parallel_num);
    void* workspace_ptr = buffer_manager.WorkspacePtr();
    size_t workspace_size = buffer_manager.WorkspaceBytes();

    CHECK_GE(tmp_buffer->shape().elem_cnt(), buffer_manager.TotalBufferSize());
    int64_t hash_capacity = num_ids;
    UniqueKeys(ctx->stream(), num_ids, hash_capacity, ids->dptr<K>(), column_ids->dptr<IDX>(),
               num_unique_ids->mut_dptr<IDX>(), ids_reverse_idx->mut_dptr<IDX>(),
               buffer_manager.UniqueIdsPtr(), buffer_manager.UniqueColumnIdsPtr(),
               reinterpret_cast<char*>(workspace_ptr), buffer_manager.WorkspaceBytes());
    // partition
    OF_CUDA_CHECK(hipMemcpyAsync(host_num_unique_ids, num_unique_ids->mut_dptr(), sizeof(IDX),
                                  hipMemcpyDefault, hip_stream));
    CHECK_JUST(ctx->stream()->Sync());

    LOG(INFO) << "rank " << parallel_id << " num_unique_ids " << *host_num_unique_ids;
    K* partitioned_unique_ids = buffer_manager.PartitionedUniqueIdsPtr();
    IDX* partitioned_num_unique_ids = buffer_manager.PartitionedNumUniqueIdsPtr();
    K* received_unique_ids = buffer_manager.ReceivedUniqueIdsPtr();
    IDX* partitioned_column_ids = buffer_manager.PartitionedColumnIdsPtr();
    IDX* received_column_ids = buffer_manager.ReceivedColumnIdsPtr();
    /*
    num_unique_ids_matrix(parallel_num * parallel_num):
           partion0   partion1
    rank0
    rank1
    */
    IDX* received_num_unique_ids_matrix = num_unique_ids_matrix->mut_dptr<IDX>();
    Partition(ctx->stream(), num_ids, host_num_unique_ids[0], parallel_num,
              buffer_manager.UniqueIdsPtr(), buffer_manager.UniqueColumnIdsPtr(),
              partitioned_unique_ids, partitioned_column_ids, partitioned_num_unique_ids,
              partition_index->mut_dptr<IDX>());

    // allgather count
    ncclComm_t comm = nccl_comm->comm();
    OF_NCCL_CHECK(ncclAllGather(reinterpret_cast<const void*>(partitioned_num_unique_ids),
                                reinterpret_cast<void*>(received_num_unique_ids_matrix),
                                parallel_num, GetNcclDataType(cur_rank_num_unique_ids->data_type()),
                                comm, hip_stream));
    IDX* host_num_unique_ids_matrix = host_num_unique_ids;
    OF_CUDA_CHECK(hipMemcpyAsync(host_num_unique_ids_matrix, received_num_unique_ids_matrix,
                                  parallel_num * parallel_num * sizeof(IDX), hipMemcpyDefault,
                                  hip_stream));
    CHECK_JUST(ctx->stream()->Sync());
    // send recv unique ids
    int64_t recv_offset = 0;
    OF_NCCL_CHECK(ncclGroupStart());
    for (int64_t j = 0; j < parallel_num; ++j) {
      const int64_t need_send_elem_cnt = host_num_unique_ids_matrix[parallel_id * parallel_num + j];
      const int64_t need_recv_elem_cnt = host_num_unique_ids_matrix[j * parallel_num + parallel_id];
      OF_NCCL_CHECK(ncclSend(reinterpret_cast<const void*>(partitioned_unique_ids + j * num_ids),
                             need_send_elem_cnt, GetNcclDataType(ids->data_type()), j, comm,
                             hip_stream));
      OF_NCCL_CHECK(ncclRecv(reinterpret_cast<void*>(received_unique_ids + recv_offset),
                             need_recv_elem_cnt, GetNcclDataType(ids->data_type()), j, comm,
                             hip_stream));
      recv_offset += need_recv_elem_cnt;
    }
    OF_NCCL_CHECK(ncclGroupEnd());
    // send recv column_ids
    int64_t slot_recv_offset = 0;
    OF_NCCL_CHECK(ncclGroupStart());
    for (int64_t j = 0; j < parallel_num; ++j) {
      const int64_t need_send_elem_cnt = host_num_unique_ids_matrix[parallel_id * parallel_num + j];
      const int64_t need_recv_elem_cnt = host_num_unique_ids_matrix[j * parallel_num + parallel_id];
      OF_NCCL_CHECK(ncclSend(reinterpret_cast<const void*>(partitioned_column_ids + j * num_ids),
                             need_send_elem_cnt, GetNcclDataType(column_ids->data_type()), j, comm,
                             hip_stream));
      OF_NCCL_CHECK(ncclRecv(reinterpret_cast<void*>(received_column_ids + slot_recv_offset),
                             need_recv_elem_cnt, GetNcclDataType(column_ids->data_type()), j, comm,
                             hip_stream));
      slot_recv_offset += need_recv_elem_cnt;
    }
    OF_NCCL_CHECK(ncclGroupEnd());

    hash_capacity = num_ids * parallel_num;
    UniqueKeys(ctx->stream(), recv_offset, hash_capacity, received_unique_ids, received_column_ids,
               cur_rank_num_unique_ids->mut_dptr<IDX>(), cur_rank_reverse_idx->mut_dptr<IDX>(),
               cur_rank_unique_ids->mut_dptr<K>(), cur_rank_column_ids->mut_dptr<IDX>(),
               reinterpret_cast<char*>(workspace_ptr), buffer_manager.WorkspaceBytes());
    if (ParseBooleanFromEnv("DEBUG_SHUFFLE", false)) { DebugIdShuffle<K, IDX>(ctx); }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename K, typename IDX>
user_op::InferTmpSizeFn GenIdShuffleInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    const user_op::TensorDesc& ids = ctx->InputTensorDesc("ids", 0);
    IdShuffleTmpBufferManager<K, IDX> buffer_manager(nullptr, ids.shape().elem_cnt(),
                                                     ctx->parallel_desc().parallel_num());
    return buffer_manager.TotalBufferSize();
  };
}

#define REGISTER_CUDA_ID_SHUFFLE_KERNEL(k_dtype, idx_dtype)                                \
  REGISTER_USER_KERNEL("id_shuffle")                                                       \
      .SetCreateFn<IdShuffleKernel<k_dtype, idx_dtype>>()                                  \
      .SetIsMatchedHob(                                                                    \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                  \
          && (user_op::HobDataType("ids", 0) == GetDataType<k_dtype>::value)               \
          && (user_op::HobDataType("num_unique_ids", 0) == GetDataType<idx_dtype>::value)) \
      .SetInferTmpSizeFn(GenIdShuffleInferTmpSizeFn<k_dtype, idx_dtype>());

// REGISTER_CUDA_ID_SHUFFLE_KERNEL(int32_t, int32_t)
REGISTER_CUDA_ID_SHUFFLE_KERNEL(int64_t, int32_t)

template<typename T, typename IDX>
class EmbeddingShuffleKernel final : public user_op::OpKernel {
 public:
  EmbeddingShuffleKernel() = default;
  ~EmbeddingShuffleKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<NcclKernelCommState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* nccl_comm = dynamic_cast<NcclKernelCommState*>(state);
    CHECK(nccl_comm != nullptr);
    ncclComm_t comm = nccl_comm->comm();

    const user_op::Tensor* cur_rank_embeddings =
        ctx->Tensor4ArgNameAndIndex("cur_rank_embeddings", 0);
    const user_op::Tensor* cur_rank_reverse_idx =
        ctx->Tensor4ArgNameAndIndex("cur_rank_reverse_idx", 0);
    const user_op::Tensor* ids_reverse_idx = ctx->Tensor4ArgNameAndIndex("ids_reverse_idx", 0);
    const user_op::Tensor* num_unique_ids_matrix =
        ctx->Tensor4ArgNameAndIndex("num_unique_ids_matrix", 0);
    const user_op::Tensor* partition_index = ctx->Tensor4ArgNameAndIndex("partition_index", 0);
    user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t num_ids = ids_reverse_idx->shape().elem_cnt();
    const int64_t embedding_size = embeddings->shape().elem_cnt() / num_ids;
    CHECK_EQ(embedding_size, ParseIntegerFromEnv("EMBEDDING_SIZE", 128));
    hipStream_t hip_stream = ctx->stream()->As<ep::CudaStream>()->hip_stream();

    IDX* host_num_unique_ids_matrix = reinterpret_cast<IDX*>(nccl_comm->HostNumUniqueIdsMatrix());
    OF_CUDA_CHECK(hipMemcpyAsync(host_num_unique_ids_matrix, num_unique_ids_matrix->dptr(),
                                  parallel_num * parallel_num * sizeof(IDX), hipMemcpyDefault,
                                  hip_stream));
    CHECK_JUST(ctx->stream()->Sync());
    int64_t cur_rank_num_ids = 0;
    for (int64_t i = 0; i < parallel_num; ++i) {
      cur_rank_num_ids += host_num_unique_ids_matrix[i * parallel_num + parallel_id];
    }

    LOG(INFO) << "parallel_id " << parallel_id << " cur_rank_num_ids before second unique "
              << cur_rank_num_ids;
    size_t reverse_cur_rank_embeddings_size =
        GetCudaAlignedSize(cur_rank_num_ids * embedding_size * sizeof(T));
    size_t recv_unique_embeddings_size =
        GetCudaAlignedSize(embeddings->shape().elem_cnt() * sizeof(T));
    T* reverse_cur_rank_embeddings = reinterpret_cast<T*>(tmp_buffer->mut_dptr());
    T* recv_unique_embeddings =
        reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + reverse_cur_rank_embeddings_size);
    T* reverse_partition_embeddings =
        reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + reverse_cur_rank_embeddings_size
                             + recv_unique_embeddings_size);

    GatherKernelUtilImpl<DeviceType::kCUDA, T, IDX>::Forward(
        ctx->stream(), cur_rank_reverse_idx->dptr<IDX>(), cur_rank_num_ids,
        cur_rank_embeddings->dptr<T>(),
        Shape({1, cur_rank_embeddings->shape().elem_cnt() / embedding_size, embedding_size}),
        reverse_cur_rank_embeddings, 0);

    int64_t send_offset = 0;
    int64_t recv_offset = 0;
    OF_NCCL_CHECK(ncclGroupStart());
    for (int64_t i = 0; i < parallel_num; ++i) {
      const int64_t need_send_elem_cnt =
          host_num_unique_ids_matrix[i * parallel_num + parallel_id] * embedding_size;
      const int64_t need_recv_elem_cnt =
          host_num_unique_ids_matrix[parallel_id * parallel_num + i] * embedding_size;
      OF_NCCL_CHECK(
          ncclSend(reinterpret_cast<const void*>(reverse_cur_rank_embeddings + send_offset),
                   need_send_elem_cnt, GetNcclDataType(cur_rank_embeddings->data_type()), i, comm,
                   hip_stream));
      OF_NCCL_CHECK(ncclRecv(reinterpret_cast<void*>(recv_unique_embeddings + recv_offset),
                             need_recv_elem_cnt, GetNcclDataType(cur_rank_embeddings->data_type()),
                             i, comm, hip_stream));
      send_offset += need_send_elem_cnt;
      recv_offset += need_recv_elem_cnt;
    }
    OF_NCCL_CHECK(ncclGroupEnd());

    int64_t offset = 0;
    for (int64_t i = 0; i < parallel_num; ++i) {
      int64_t elem_cnt =
          host_num_unique_ids_matrix[parallel_id * parallel_num + i] * embedding_size;
      ReversePartitionEmbeddingKernel<T, IDX>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, hip_stream>>>(
              elem_cnt, embedding_size, partition_index->dptr<IDX>() + i * num_ids,
              recv_unique_embeddings + offset, reverse_partition_embeddings);
      offset += elem_cnt;
    }
    GatherKernelUtilImpl<DeviceType::kCUDA, T, IDX>::Forward(
        ctx->stream(), ids_reverse_idx->dptr<IDX>(), ids_reverse_idx->shape().elem_cnt(),
        reverse_partition_embeddings, Shape({1, num_ids, embedding_size}),
        embeddings->mut_dptr<T>(), 0);

    if (ParseBooleanFromEnv("DEBUG_SHUFFLE", false)) { DebugEmbeddingShuffle<T>(ctx); }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T, typename IDX>
user_op::InferTmpSizeFn GenEmbeddingShuffleInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    const user_op::TensorDesc& cur_rank_embeddings = ctx->InputTensorDesc("cur_rank_embeddings", 0);
    const user_op::TensorDesc& embeddings = ctx->InputTensorDesc("embeddings", 0);
    size_t reverse_cur_rank_embeddings_size =
        GetCudaAlignedSize(cur_rank_embeddings.shape().elem_cnt() * sizeof(T));
    size_t recv_unique_embeddings = GetCudaAlignedSize(embeddings.shape().elem_cnt() * sizeof(T));
    return reverse_cur_rank_embeddings_size + 2 * recv_unique_embeddings;
  };
}

#define REGISTER_CUDA_EMBEDDING_SHUFFLE_KERNEL(t_dtype, idx_dtype)                              \
  REGISTER_USER_KERNEL("embedding_shuffle")                                                     \
      .SetCreateFn<EmbeddingShuffleKernel<t_dtype, idx_dtype>>()                                \
      .SetIsMatchedHob(                                                                         \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                       \
          && (user_op::HobDataType("cur_rank_reverse_idx", 0) == GetDataType<idx_dtype>::value) \
          && (user_op::HobDataType("embeddings", 0) == GetDataType<t_dtype>::value))            \
      .SetInferTmpSizeFn(GenEmbeddingShuffleInferTmpSizeFn<t_dtype, idx_dtype>());

REGISTER_CUDA_EMBEDDING_SHUFFLE_KERNEL(float, int32_t)
REGISTER_CUDA_EMBEDDING_SHUFFLE_KERNEL(float16, int32_t)

namespace {

template<typename T, typename IDX>
typename std::enable_if<!std::is_same<T, float16>::value, void>::type UnsortedSegmentSum(
    ep::Stream* stream, const IDX* idx, const T* data, int64_t num_ids, int64_t embedding_size,
    T* out, float* tmp_buffer) {
  Memset<DeviceType::kCUDA>(stream, out, 0, num_ids * embedding_size * sizeof(T));
  UnsortedSegmentSumKernelUtil<DeviceType::kCUDA, T, IDX, T>::UnsortedSegmentSum(
      stream, idx, data, num_ids, num_ids, 1, embedding_size, 0, out);
}

template<typename T, typename IDX>
typename std::enable_if<std::is_same<T, float16>::value, void>::type UnsortedSegmentSum(
    ep::Stream* stream, const IDX* idx, const T* data, int64_t num_ids, int64_t embedding_size,
    T* out, float* tmp_buffer) {
  Memset<DeviceType::kCUDA>(stream, tmp_buffer, 0, num_ids * embedding_size * sizeof(float));
  Memset<DeviceType::kCUDA>(stream, out, 0, num_ids * embedding_size * sizeof(T));
  UnsortedSegmentSumKernelUtil<DeviceType::kCUDA, float, IDX, T>::UnsortedSegmentSum(
      stream, idx, data, num_ids, num_ids, 1, embedding_size, 0, tmp_buffer);
  auto f2h = ep::primitive::NewPrimitive<ep::primitive::CastFactory>(
      DeviceType::kCUDA, DataType::kFloat, DataType::kFloat16);
  CHECK(f2h);
  f2h->Launch(stream, tmp_buffer, out, num_ids * embedding_size);
}

}  // namespace

template<typename T, typename IDX>
class EmbeddingGradientShuffleKernel final : public user_op::OpKernel {
 public:
  EmbeddingGradientShuffleKernel() = default;
  ~EmbeddingGradientShuffleKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    return std::make_shared<NcclKernelCommState>(ctx);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    auto* nccl_comm = dynamic_cast<NcclKernelCommState*>(state);
    CHECK(nccl_comm != nullptr);
    ncclComm_t comm = nccl_comm->comm();

    const user_op::Tensor* embedding_diff = ctx->Tensor4ArgNameAndIndex("embedding_diff", 0);
    const user_op::Tensor* cur_rank_reverse_idx =
        ctx->Tensor4ArgNameAndIndex("cur_rank_reverse_idx", 0);
    const user_op::Tensor* ids_reverse_idx = ctx->Tensor4ArgNameAndIndex("ids_reverse_idx", 0);
    const user_op::Tensor* num_unique_ids_matrix =
        ctx->Tensor4ArgNameAndIndex("num_unique_ids_matrix", 0);
    const user_op::Tensor* partition_index = ctx->Tensor4ArgNameAndIndex("partition_index", 0);
    user_op::Tensor* cur_rank_unique_embedding_diff =
        ctx->Tensor4ArgNameAndIndex("cur_rank_unique_embedding_diff", 0);
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    const int64_t parallel_id = ctx->parallel_ctx().parallel_id();
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t num_ids = ids_reverse_idx->shape().elem_cnt();
    const int64_t embedding_size = embedding_diff->shape().elem_cnt() / num_ids;
    hipStream_t hip_stream = ctx->stream()->As<ep::CudaStream>()->hip_stream();
    IDX* host_num_unique_ids_matrix = reinterpret_cast<IDX*>(nccl_comm->HostNumUniqueIdsMatrix());
    OF_CUDA_CHECK(hipMemcpyAsync(host_num_unique_ids_matrix, num_unique_ids_matrix->dptr(),
                                  parallel_num * parallel_num * sizeof(IDX), hipMemcpyDefault,
                                  hip_stream));
    CHECK_JUST(ctx->stream()->Sync());
    int64_t cur_rank_num_ids = 0;
    for (int64_t i = 0; i < parallel_num; ++i) {
      cur_rank_num_ids += host_num_unique_ids_matrix[i * parallel_num + parallel_id];
    }

    size_t unique_diff_size = GetCudaAlignedSize(embedding_diff->shape().elem_cnt() * sizeof(T));
    size_t recv_unique_embeddings_size =
        GetCudaAlignedSize(cur_rank_unique_embedding_diff->shape().elem_cnt() * sizeof(T));
    T* unique_diff_ptr = reinterpret_cast<T*>(tmp_buffer->mut_dptr());
    T* recv_embeddings_diff = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + unique_diff_size);
    T* partitioned_embedding = reinterpret_cast<T*>(tmp_buffer->mut_dptr<char>() + unique_diff_size
                                                    + recv_unique_embeddings_size);
    size_t unsorted_segment_sum_tmp_buffer_size =
        (embedding_diff->data_type() == DataType::kFloat16)
            ? GetCudaAlignedSize(cur_rank_unique_embedding_diff->shape().elem_cnt() * sizeof(float))
            : 0;
    float* unsorted_segment_sum_tmp_buffer = reinterpret_cast<float*>(
        tmp_buffer->mut_dptr<char>() + 2 * unique_diff_size + recv_unique_embeddings_size);

    UnsortedSegmentSum<T, IDX>(ctx->stream(), ids_reverse_idx->dptr<IDX>(),
                               embedding_diff->dptr<T>(), num_ids, embedding_size, unique_diff_ptr,
                               unsorted_segment_sum_tmp_buffer);
    int64_t offset = 0;
    for (int64_t i = 0; i < parallel_num; ++i) {
      int64_t elem_cnt =
          host_num_unique_ids_matrix[parallel_id * parallel_num + i] * embedding_size;
      PartitionEmbeddingKernel<T, IDX>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, hip_stream>>>(
              elem_cnt, embedding_size, partition_index->dptr<IDX>() + i * num_ids, unique_diff_ptr,
              partitioned_embedding + offset);
      offset += elem_cnt;
    }
    int64_t send_offset = 0;
    int64_t recv_offset = 0;
    OF_NCCL_CHECK(ncclGroupStart());
    for (int64_t i = 0; i < parallel_num; ++i) {
      const int64_t need_send_elem_cnt =
          host_num_unique_ids_matrix[parallel_id * parallel_num + i] * embedding_size;
      const int64_t need_recv_elem_cnt =
          host_num_unique_ids_matrix[i * parallel_num + parallel_id] * embedding_size;
      OF_NCCL_CHECK(ncclSend(reinterpret_cast<const void*>(partitioned_embedding + send_offset),
                             need_send_elem_cnt, GetNcclDataType(embedding_diff->data_type()), i,
                             comm, hip_stream));
      OF_NCCL_CHECK(ncclRecv(reinterpret_cast<void*>(recv_embeddings_diff + recv_offset),
                             need_recv_elem_cnt, GetNcclDataType(embedding_diff->data_type()), i,
                             comm, hip_stream));
      send_offset += need_send_elem_cnt;
      recv_offset += need_recv_elem_cnt;
    }
    OF_NCCL_CHECK(ncclGroupEnd());

    UnsortedSegmentSum<T, IDX>(ctx->stream(), cur_rank_reverse_idx->dptr<IDX>(),
                               recv_embeddings_diff, cur_rank_num_ids, embedding_size,
                               cur_rank_unique_embedding_diff->mut_dptr<T>(),
                               unsorted_segment_sum_tmp_buffer);

    if (ParseBooleanFromEnv("DEBUG_SHUFFLE", false)) { DebugEmbeddingGradientShuffle<T>(ctx); }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T, typename IDX>
user_op::InferTmpSizeFn GenEmbeddingGradientShuffleInferTmpSizeFn() {
  return [](user_op::InferContext* ctx) {
    const user_op::TensorDesc& embedding_diff = ctx->InputTensorDesc("embedding_diff", 0);
    const user_op::TensorDesc& cur_rank_unique_embedding_diff =
        ctx->InputTensorDesc("cur_rank_unique_embedding_diff", 0);
    size_t unique_embedding_diff_size = embedding_diff.shape().elem_cnt() * sizeof(T);
    size_t cur_rank_unique_embedding_diff_size =
        cur_rank_unique_embedding_diff.shape().elem_cnt() * sizeof(T);
    size_t unsorted_segment_sum_tmp_buffer_size =
        (embedding_diff.data_type() == DataType::kFloat16)
            ? GetCudaAlignedSize(cur_rank_unique_embedding_diff.shape().elem_cnt() * sizeof(float))
            : 0;
    return 2 * unique_embedding_diff_size + cur_rank_unique_embedding_diff_size
           + unsorted_segment_sum_tmp_buffer_size;
  };
}

#define REGISTER_CUDA_EMBEDDING_GRADIENT_SHUFFLE_KERNEL(t_dtype, idx_dtype)                \
  REGISTER_USER_KERNEL("embedding_gradient_shuffle")                                       \
      .SetCreateFn<EmbeddingGradientShuffleKernel<t_dtype, idx_dtype>>()                   \
      .SetIsMatchedHob(                                                                    \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                  \
          && (user_op::HobDataType("ids_reverse_idx", 0) == GetDataType<idx_dtype>::value) \
          && (user_op::HobDataType("embedding_diff", 0) == GetDataType<t_dtype>::value))   \
      .SetInferTmpSizeFn(GenEmbeddingGradientShuffleInferTmpSizeFn<t_dtype, idx_dtype>());

REGISTER_CUDA_EMBEDDING_GRADIENT_SHUFFLE_KERNEL(float, int32_t)
REGISTER_CUDA_EMBEDDING_GRADIENT_SHUFFLE_KERNEL(float16, int32_t)

}  // namespace oneflow
