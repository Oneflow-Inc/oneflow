#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/primitive/include/permute.h"
#include "oneflow/core/primitive/common/permute.h"
#include "oneflow/core/stream/cuda_stream_context.h"
#include "oneflow/core/primitive/cuda/cuda_graph_support.h"
#include <hip/hip_runtime.h>

namespace oneflow {

namespace primitive {

namespace permute_internal {

namespace {

template<size_t num_dims, size_t movement_size, typename IndexType>
__global__ void PermuteKernel(PermuteKernelParams<num_dims, IndexType> params) {
  using T = typename std::aligned_storage<movement_size, movement_size>::type;
  const T* src = reinterpret_cast<const T*>(params.src);
  T* dst = reinterpret_cast<T*>(params.dst);
  CUDA_1D_KERNEL_LOOP_T(IndexType, i, params.count) {
    IndexType src_index[num_dims];
    IndexType dst_index[num_dims];
    params.dst_index_helper.OffsetToNdIndex(i, dst_index);
#pragma unroll
    for (size_t dim = 0; dim < num_dims; ++dim) {
      src_index[params.permutation[dim]] = dst_index[dim];
    }
    IndexType src_offset = params.src_index_helper.NdIndexToOffset(src_index);
    dst[i] = src[src_offset];
  }
}

template<size_t num_dims, size_t movement_size, typename IndexType>
void LaunchKernel(StreamContext* stream_ctx, PermuteKernelParams<num_dims, IndexType> params) {
  hipStream_t hip_stream =
      CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();
  PermuteKernel<num_dims, movement_size, IndexType>
      <<<BlocksNum4ThreadsNum(params.count), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params);
}

class PermuteImpl : public Permute, public CudaGraphSupport {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteImpl);
  PermuteImpl() = default;
  ~PermuteImpl() override = default;

  void Launch(StreamContext* stream_ctx, DataType data_type, size_t num_dims,
              const int64_t* src_dims, const void* src, const int* permutation,
              void* dst) override {
    SimplifyThenLaunch(stream_ctx, data_type, num_dims, src_dims, src, permutation, dst);
  }
};

class PermuteFactoryImpl : public PermuteFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteFactoryImpl);
  PermuteFactoryImpl() = default;
  ~PermuteFactoryImpl() override = default;

  std::unique_ptr<Permute> New(size_t max_num_dims) override {
    if (max_num_dims <= kMaxNumDims) {
      return std::unique_ptr<Permute>(new PermuteImpl());
    } else {
      return nullptr;
    }
  }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kGPU, PermuteFactory, PermuteFactoryImpl);

}  // namespace

}  // namespace permute_internal

}  // namespace primitive

}  // namespace oneflow
