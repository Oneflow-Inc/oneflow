#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/primitive/include/permute.h"
#include "oneflow/core/primitive/common/permute.h"
#include "oneflow/core/stream/cuda_stream_context.h"
#include "oneflow/core/primitive/cuda/cuda_graph_support.h"
#include <hip/hip_runtime.h>

namespace oneflow {

namespace primitive {

namespace permute_internal {

namespace {


template<size_t num_dims, size_t movement_size, typename IndexType>
__global__ void PermuteKernel(PermuteKernelParams<num_dims, IndexType> params) {
  using T = typename std::aligned_storage<movement_size, movement_size>::type;
  const T* src = reinterpret_cast<const T*>(params.src);
  T* dst = reinterpret_cast<T*>(params.dst);
  IndexType src_index[num_dims];
  IndexType dst_index[num_dims];
  CUDA_1D_KERNEL_LOOP_T(IndexType, i, params.count) {
    params.dst_index_helper.OffsetToNdIndex(i, dst_index);
#pragma unroll
    for (size_t dim = 0; dim < num_dims; ++dim) {
      src_index[params.permutation[dim]] = dst_index[dim];
    }
    IndexType src_offset = params.src_index_helper.NdIndexToOffset(src_index);
    dst[i] = src[src_offset];
  }
}

constexpr int32_t TILE_SIZE = 32; // float tile size. 
constexpr int32_t HALF_TILE_SIZE = 64; // cause float16 is half of float32, we need to double tilesize for half kernel. 
constexpr int32_t KBLOCKROWS = 8;

// (B, X, Y) -> (B, Y, X), refer from
// https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
template<size_t num_dims, size_t movement_size, size_t tile_size, typename IndexType>
__global__ void BatchPermuteKernel(PermuteKernelParams<num_dims, IndexType> params, IndexType H,
                                   IndexType W, IndexType dh, IndexType dw, int32_t grid_size) {
  using T = typename std::aligned_storage<movement_size, movement_size>::type;
  __shared__ T tile[tile_size][tile_size + 1];  // To avoid bank conflict.
  const T* src = reinterpret_cast<const T*>(params.src);
  T* dst = reinterpret_cast<T*>(params.dst);
  IndexType dh_mul_dw = dh * dw; 
  for (int i = blockIdx.x, step = gridDim.x; i < grid_size; i += step) {
    const IndexType n = i / dh_mul_dw;  // the index of batch.
    const IndexType k = i - n * dh_mul_dw;  // equal to i % (dh*dw). the flatten index of tile in a batch. 

    const IndexType r = k / dw;  // the row index of tile in a batch.
    const IndexType c = k - r * dw; // equal to k % dw. the col index of tile in a batch.
    const IndexType offset = n * H * W;
    int x = c * tile_size + threadIdx.x;
    int y = r * tile_size + threadIdx.y;
    if (x < W) {
#pragma unroll
      // each thread process 4 elements.
      for (int i = 0; threadIdx.y + i < tile_size && y + i < H; i += KBLOCKROWS) {
        tile[threadIdx.y + i][threadIdx.x] = src[offset + (y + i) * W + x];
      }
    }
    __syncthreads();
    x = r * tile_size + threadIdx.x;
    y = c * tile_size + threadIdx.y;
    if (x < H) {
#pragma unroll
      for (int i = 0; threadIdx.y + i < tile_size && y + i < W; i += KBLOCKROWS) {
        dst[offset + (y + i) * H + x] = tile[threadIdx.x][threadIdx.y + i];
      }
    }
    __syncthreads();
  }
}

/*
Here is a half2 version of Batch Permute. 
When the H W can be divided by 2. we can read data use half2, and write back as half. 
We design a union structure to store half and half2 share memory. 
Actually here shared memory size is Half[64][66]. 
*/
template<size_t num_dims, size_t movement_size, size_t tile_size, typename IndexType>
__global__ void BatchPermuteHalf2Kernel(PermuteKernelParams<num_dims, IndexType> params, IndexType H,
                                   IndexType W, IndexType dh, IndexType dw, int32_t grid_size) {
  // Use union structure to process Load and Store. 
  __shared__ union {
    half tile_half[tile_size][tile_size+2]; // [64][66]
    half2 tile_half2[tile_size/2][tile_size/2+1]; // [32][33]
  } tile_mem;

  const half2* src = reinterpret_cast<const half2*>(params.src);
  half* dst = reinterpret_cast<half*>(params.dst);
  
  IndexType dh_mul_dw = dh * dw; 
  for (int i = blockIdx.x, step = gridDim.x; i < grid_size; i += step) {
    const IndexType n = i / dh_mul_dw;  // the index of batch.
    const IndexType k = i - n * dh_mul_dw;  // equal to i%(dh*dw). the flatten index of tile in a batch. TODO! optimize it!

    const IndexType r = k / dw;  // the row index of tile in a batch.
    const IndexType c = k - r * dw; // equal to k % dw. the col index of tile in a batch.
    const IndexType offset = n * H * W;
    int x = c * tile_size + threadIdx.x*2; // cause each thread process a half2 element, we need to multiply 2 for threadIdx.x. 
    int y = r * tile_size + threadIdx.y;
    if (x < W) {
#pragma unroll
      // each thread process 4 elements.
      for (int i = 0; threadIdx.y + i < tile_size && y + i < H; i += KBLOCKROWS) {
        // each thread load a half2. 
        tile_mem.tile_half2[threadIdx.y + i][threadIdx.x] = src[(offset + (y + i) * W + x)/2];
      }
    }
    __syncthreads();
    x = r * tile_size + threadIdx.x*2; // cause each thread process a half2 element, we need to multiply 2 for threadIdx.x. 
    y = c * tile_size + threadIdx.y; 
    if (x < H) {
#pragma unroll
      for (int i = 0; threadIdx.y + i < tile_size && y + i < W; i += KBLOCKROWS) {
        /*
        When write back as column, it cannot be stored as half2 directly. 
        So we split as 2 half elements, and write back separately. 
        */
        dst[offset + (y + i) * H + x] = tile_mem.tile_half[threadIdx.x*2][threadIdx.y + i]; 
        dst[offset + (y + i) * H + x + 1] = tile_mem.tile_half[threadIdx.x*2 + 1][threadIdx.y + i]; 
      }
    }
    __syncthreads();
  }
}


template<size_t num_dims, size_t movement_size, size_t tile_size, typename IndexType>
void LaunchBatchPermuteKernel(StreamContext* stream_ctx,
                              PermuteKernelParams<num_dims, IndexType> params, IndexType& n,
                              IndexType& h, IndexType& w) {
  hipStream_t hip_stream =
      CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();

  IndexType dh = (h + tile_size - 1) / tile_size;
  IndexType dw = (w + tile_size - 1) / tile_size;

  const int32_t grid_size = n * dh * dw;
  int32_t checked_grid_size = std::min(grid_size, kCudaMaxBlocksNum); 
  if(tile_size == HALF_TILE_SIZE){
    const int32_t half2_thread = tile_size / 2; // cause each thread process two half elements. 
    // Only specialized movementsize==2 to avoid using too much shared memory. 
    BatchPermuteHalf2Kernel<num_dims, 2, HALF_TILE_SIZE, IndexType>
      <<<checked_grid_size, dim3(half2_thread, KBLOCKROWS), 0, hip_stream>>>(
          params, h, w, dh, dw, grid_size);  // Set threads num as 32x8 cause each threads process 4
                                             // elements to 32x32 share memory.
  }else{
    BatchPermuteKernel<num_dims, movement_size, tile_size, IndexType>
      <<<checked_grid_size, dim3(tile_size, KBLOCKROWS), 0, hip_stream>>>(
          params, h, w, dh, dw, grid_size);  
  }
}


template<size_t tile_size, typename IndexType>
bool CheckIfGreaterThanTileSize(IndexType& h, IndexType& w) {
  // H W should be less than tile size.
  if (h < tile_size || w < tile_size) { return false; }
  return true;
}

template<size_t num_dims, size_t tile_size, typename IndexType>
bool CheckLaunchBatchPermute(PermuteKernelParams<num_dims, IndexType> params, IndexType& n,
                             IndexType& h, IndexType& w) {
  if (CheckIfGreaterThanTileSize<tile_size, IndexType>(h, w)) {
    if (n == 1) {
      return true;
    } else if (num_dims == 3 && params.permutation[2] == 1 && params.permutation[1] == 2) {
      return true;
    } else {
      return false;
    }
  }
  return false;
}

template<typename IndexType, size_t movement_size>
bool CheckUseHalf2(IndexType& h, IndexType& w) {
  if(movement_size == 2){ // movement_size = 2, means half type
    if(h % 2 == 0 && w % 2 ==0){
      // When h and w can both divided by 2, it means we can use half2 type to improve load efficiency. 
      return true; 
    }
  }
  return false;
}


template<size_t num_dims, size_t movement_size, typename IndexType>
void LaunchKernel(StreamContext* stream_ctx, PermuteKernelParams<num_dims, IndexType> params) {
  hipStream_t hip_stream =
      CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();

  if (num_dims == 2 || num_dims == 3) {
    IndexType n;
    IndexType h;
    IndexType w;
    if (num_dims == 2) {
      IndexType global_index[2];
      /*
      For example: assume dim is (4, 6), offset = 24.
      offset-1=23, convet back to NdIndex is (3, 5), cause index start from zero and we need to
      subtract 1. then we add 1 to all the NdIndex to get the actual dim.
      */
      params.src_index_helper.OffsetToNdIndex(params.count - 1, global_index);
      n = 1;
      h = global_index[0] + 1;
      w = global_index[1] + 1;
    } else {
      IndexType global_index[3];
      params.src_index_helper.OffsetToNdIndex(params.count - 1, global_index);
      n = global_index[0] + 1;
      h = global_index[1] + 1;
      w = global_index[2] + 1;
    }
    if (CheckLaunchBatchPermute<num_dims, TILE_SIZE>(params, n, h, w)) {
      if(CheckUseHalf2<IndexType, movement_size>(h, w)){
        LaunchBatchPermuteKernel<num_dims, 2, HALF_TILE_SIZE, IndexType>(stream_ctx, params, n, h, w);  
      }else{
        LaunchBatchPermuteKernel<num_dims, movement_size, TILE_SIZE, IndexType>(stream_ctx, params, n, h, w);    
      }
    } else {
      PermuteKernel<num_dims, movement_size, IndexType>
          <<<BlocksNum4ThreadsNum(params.count), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params);
    }
  } else {
    PermuteKernel<num_dims, movement_size, IndexType>
        <<<BlocksNum4ThreadsNum(params.count), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params);
  }

}

class PermuteImpl : public Permute, public CudaGraphSupport {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteImpl);
  PermuteImpl() = default;
  ~PermuteImpl() override = default;

  using Permute::Launch;
  void Launch(StreamContext* stream_ctx, DataType data_type, size_t num_dims,
              const int64_t* src_dims, const void* src, const int* permutation,
              void* dst) override {
    SimplifyThenLaunch(stream_ctx, data_type, num_dims, src_dims, src, permutation, dst);
  }
};

class PermuteFactoryImpl : public PermuteFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteFactoryImpl);
  PermuteFactoryImpl() = default;
  ~PermuteFactoryImpl() override = default;

  std::unique_ptr<Permute> New(size_t max_num_dims) override {
    if (max_num_dims <= kMaxNumDims) {
      return std::unique_ptr<Permute>(new PermuteImpl());
    } else {
      return nullptr;
    }
  }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kGPU, PermuteFactory, PermuteFactoryImpl);

}  // namespace

}  // namespace permute_internal

}  // namespace primitive

}  // namespace oneflow
