#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/primitive/include/permute.h"
#include "oneflow/core/primitive/common/permute.h"
#include "oneflow/core/stream/cuda_stream_context.h"
#include "oneflow/core/primitive/cuda/cuda_graph_support.h"
#include <hip/hip_runtime.h>

namespace oneflow {

namespace primitive {

namespace permute_internal {

namespace {


template<size_t num_dims, size_t movement_size, typename IndexType>
__global__ void PermuteKernel(PermuteKernelParams<num_dims, IndexType> params) {
  using T = typename std::aligned_storage<movement_size, movement_size>::type;
  const T* src = reinterpret_cast<const T*>(params.src);
  T* dst = reinterpret_cast<T*>(params.dst);
  IndexType src_index[num_dims];
  IndexType dst_index[num_dims];
  CUDA_1D_KERNEL_LOOP_T(IndexType, i, params.count) {
    params.dst_index_helper.OffsetToNdIndex(i, dst_index);
#pragma unroll
    for (size_t dim = 0; dim < num_dims; ++dim) {
      src_index[params.permutation[dim]] = dst_index[dim];
    }
    IndexType src_offset = params.src_index_helper.NdIndexToOffset(src_index);
    dst[i] = src[src_offset];
  }
}

// tile_size = 32 x 32
constexpr int32_t tile_size = 32;
constexpr int32_t kBlockRows = 8;

// (B, X, Y) -> (B, Y, X), refer from
// https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
// template<size_t num_dims, size_t movement_size, size_t tile_size, typename IndexType>
// __global__ void BatchPermuteKernel(PermuteKernelParams<num_dims, IndexType> params, IndexType H,
//                                    IndexType W, IndexType dh, IndexType dw, int32_t grid_size) {
//   using T = typename std::aligned_storage<movement_size, movement_size>::type;
//   __shared__ T tile[tile_size][tile_size + 1];  // To avoid bank conflict.
//   const T* src = reinterpret_cast<const T*>(params.src);
//   T* dst = reinterpret_cast<T*>(params.dst);
//   IndexType dh_mul_dw = dh * dw; 
//   for (int i = blockIdx.x, step = gridDim.x; i < grid_size; i += step) {
//     const IndexType n = i / dh_mul_dw;  // the index of batch.
//     const IndexType k = i - n * dh_mul_dw;  // equal to i%(dh*dw). the flatten index of tile in a batch. TODO! optimize it!

//     const IndexType r = k / dw;  // the row index of tile in a batch.
//     const IndexType c = k - r * dw; // equal to k% dw. the col index of tile in a batch.
//     const IndexType offset = n * H * W;
//     int x = c * tile_size + threadIdx.x;
//     int y = r * tile_size + threadIdx.y;
//     if (x < W) {
// #pragma unroll
//       // each thread process 4 elements.
//       for (int i = 0; threadIdx.y + i < tile_size && y + i < H; i += kBlockRows) {
//         tile[threadIdx.y + i][threadIdx.x] = src[offset + (y + i) * W + x];
//       }
//     }
//     __syncthreads();
//     x = r * tile_size + threadIdx.x;
//     y = c * tile_size + threadIdx.y;
//     if (x < H) {
// #pragma unroll
//       for (int i = 0; threadIdx.y + i < tile_size && y + i < W; i += kBlockRows) {
//         dst[offset + (y + i) * H + x] = tile[threadIdx.x][threadIdx.y + i];
//       }
//     }
//     __syncthreads();
//   }
// }

// template<size_t num_dims, size_t movement_size, size_t tile_size, typename IndexType>
// __global__ void BatchPermuteHalf2Kernel(PermuteKernelParams<num_dims, IndexType> params, IndexType H,
//                                    IndexType W, IndexType dh, IndexType dw, int32_t grid_size) {
//   using T = typename std::aligned_storage<movement_size, movement_size>::type;
//   __shared__ T tile[tile_size][tile_size + 1];  // To avoid bank conflict. half[32][66]; 

//   const T* src = reinterpret_cast<const T*>(params.src);
//   T* dst = reinterpret_cast<T*>(params.dst);
//   IndexType dh_mul_dw = dh * dw; 
//   for (int i = blockIdx.x, step = gridDim.x; i < grid_size; i += step) {
//     const IndexType n = i / dh_mul_dw;  // the index of batch.
//     const IndexType k = i - n * dh_mul_dw;  // equal to i%(dh*dw). the flatten index of tile in a batch. TODO! optimize it!

//     const IndexType r = k / dw;  // the row index of tile in a batch.
//     const IndexType c = k - r * dw; // equal to k% dw. the col index of tile in a batch.
//     const IndexType offset = n * H * W;
//     int x = c * tile_size + threadIdx.x*2;
//     int y = r * tile_size + threadIdx.y;
//     if (x < W) {
// #pragma unroll
//       // each thread process 4 elements.
//       for (int i = 0; threadIdx.y + i < tile_size && y + i < H; i += kBlockRows) {
//         tile[threadIdx.y + i][threadIdx.x*2] = src[offset + (y + i) * W + x];
//         tile[threadIdx.y + i][threadIdx.x*2 + 1] = src[offset + (y + i) * W + x + 1];
//       }
//     }
//     __syncthreads();
//     x = r * tile_size + threadIdx.x*2;
//     y = c * tile_size + threadIdx.y; 
//     if (x < H) {
// #pragma unroll
//       for (int i = 0; threadIdx.y + i < tile_size && y + i < W; i += kBlockRows) {
//         dst[offset + (y + i) * H + x] = tile[threadIdx.x*2][threadIdx.y + i]; 
//         dst[offset + (y + i) * H + x + 1] = tile[threadIdx.x*2 + 1][threadIdx.y + i]; 
//       }
//     }
//     __syncthreads();
//   }
// }


template<size_t num_dims, size_t movement_size, size_t tile_size, typename IndexType>
__global__ void BatchPermuteHalf2Kernel(PermuteKernelParams<num_dims, IndexType> params, IndexType H,
                                   IndexType W, IndexType dh, IndexType dw, int32_t grid_size) {
  using T = typename std::aligned_storage<movement_size, movement_size>::type;
  
  __shared__ union {
    half tile_half[tile_size][tile_size+2]; // [64][66]
    half2 tile_half2[tile_size/2][tile_size/2+1]; // [32][33]
  } tile_mem;

  const half2* src = reinterpret_cast<const half2*>(params.src);
  half* dst = reinterpret_cast<half*>(params.dst);
  
  IndexType dh_mul_dw = dh * dw; 
  for (int i = blockIdx.x, step = gridDim.x; i < grid_size; i += step) {
    const IndexType n = i / dh_mul_dw;  // the index of batch.
    const IndexType k = i - n * dh_mul_dw;  // equal to i%(dh*dw). the flatten index of tile in a batch. TODO! optimize it!

    const IndexType r = k / dw;  // the row index of tile in a batch.
    const IndexType c = k - r * dw; // equal to k% dw. the col index of tile in a batch.
    const IndexType offset = n * H * W;
    int x = c * tile_size + threadIdx.x*2;
    int y = r * tile_size + threadIdx.y;
    if (x < W) {
#pragma unroll
      // each thread process 4 elements.
      for (int i = 0; threadIdx.y + i < tile_size && y + i < H; i += kBlockRows) {
        tile_mem.tile_half2[threadIdx.y + i][threadIdx.x] = src[(offset + (y + i) * W + x)/2];
      }
    }
    __syncthreads();

    x = r * tile_size + threadIdx.x*2;
    y = c * tile_size + threadIdx.y; 
    if (x < H) {
#pragma unroll
      for (int i = 0; threadIdx.y + i < tile_size && y + i < W; i += kBlockRows) {
        dst[offset + (y + i) * H + x] = tile_mem.tile_half[threadIdx.x*2][threadIdx.y + i]; 
        dst[offset + (y + i) * H + x + 1] = tile_mem.tile_half[threadIdx.x*2 + 1][threadIdx.y + i]; 
      }
    }
    __syncthreads();
  }
}


template<size_t num_dims, size_t movement_size, size_t tile_size, typename IndexType>
void LaunchBatchPermuteKernel(StreamContext* stream_ctx,
                              PermuteKernelParams<num_dims, IndexType> params, IndexType& n,
                              IndexType& h, IndexType& w) {
  hipStream_t hip_stream =
      CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();

  IndexType dh = (h + tile_size - 1) / tile_size;
  IndexType dw = (w + tile_size - 1) / tile_size;
  printf("Dh is: %d \n", dh);
  printf("Dw is: %d \n", dw);

  const int32_t grid_size = n * dh * dw;
  int32_t checked_grid_size = std::min(grid_size, kCudaMaxBlocksNum); 
  printf("Use Batch Permute Kernel!!! \n");
  printf("Checked grid size is: %d \n", checked_grid_size);
  printf("kCudaMaxBlocks is: %d \n", kCudaMaxBlocksNum);

  // BatchPermuteKernel<num_dims, movement_size, tile_size, IndexType>
  //     <<<checked_grid_size, dim3(tile_size, kBlockRows), 0, hip_stream>>>(
  //         params, h, w, dh, dw, grid_size);  // Set threads num as 32x8 cause each threads process 4
  //                                            // elements to 32x32 share memory.
  
  int32_t half2_thread = tile_size / 2; 
  printf("Half 2 thread num is: %d \n", half2_thread); 
  BatchPermuteHalf2Kernel<num_dims, 2, tile_size, IndexType>
      <<<checked_grid_size, dim3(half2_thread, kBlockRows), 0, hip_stream>>>(
          params, h, w, dh, dw, grid_size);  // Set threads num as 32x8 cause each threads process 4
                                             // elements to 32x32 share memory.
}


template<size_t tile_size, typename IndexType>
bool CheckIfGreaterThanTileSize(IndexType& h, IndexType& w) {
  // H W should be less than tile size.
  if (h < tile_size || w < tile_size) { return false; }
  return true;
}

template<size_t num_dims, size_t tile_size, typename IndexType>
bool CheckLaunchBatchPermute(PermuteKernelParams<num_dims, IndexType> params, IndexType& n,
                             IndexType& h, IndexType& w) {
  if (CheckIfGreaterThanTileSize<tile_size, IndexType>(h, w)) {
    if (n == 1) {
      return true;
    } else if (num_dims == 3 && params.permutation[2] == 1 && params.permutation[1] == 2) {
      return true;
    } else {
      return false;
    }
  }
  return false;
}

template<typename IndexType, size_t movement_size>
bool CheckUseHalf2(IndexType& h, IndexType& w) {
  if(movement_size == 2){ // movement_size = 2, means half type
    if(h % 2 == 0 && w % 2 ==0){
      return true; 
    }
  }
  return false;
}


template<size_t num_dims, size_t movement_size, typename IndexType>
void LaunchKernel(StreamContext* stream_ctx, PermuteKernelParams<num_dims, IndexType> params) {
  hipStream_t hip_stream =
      CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();

  if (num_dims == 2 || num_dims == 3) {
    IndexType n;
    IndexType h;
    IndexType w;
    if (num_dims == 2) {
      IndexType global_index[2];
      params.src_index_helper.OffsetToNdIndex(params.count - 1, global_index);
      /*
      For example: assume dim is (4, 6), offset = 24.
      offset-1=23, convet back to NdIndex is (3, 5), cause index start from zero and we need to
      subtract 1. then we add 1 to all the NdIndex to get the actual dim.
      */
      n = 1;
      h = global_index[0] + 1;
      w = global_index[1] + 1;
    } else {
      IndexType global_index[3];
      params.src_index_helper.OffsetToNdIndex(params.count - 1, global_index);
      n = global_index[0] + 1;
      h = global_index[1] + 1;
      w = global_index[2] + 1;
    }
    printf("n is: %d \n", n);
    printf("h is: %d \n", h);
    printf("w is: %d \n", w);
    if (CheckLaunchBatchPermute<num_dims, tile_size>(params, n, h, w)) {
      if(CheckUseHalf2<IndexType, movement_size>(h, w)){
        // use half2. 
        printf("Here use half! \n"); 
        LaunchBatchPermuteKernel<num_dims, movement_size, 64, IndexType>(stream_ctx, params, n, h, w);  
      }else{
        LaunchBatchPermuteKernel<num_dims, movement_size, tile_size, IndexType>(stream_ctx, params, n,
                                                                                      h, w);    
      }
    } else {
      PermuteKernel<num_dims, movement_size, IndexType>
          <<<BlocksNum4ThreadsNum(params.count), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params);
    }
  } else {
    PermuteKernel<num_dims, movement_size, IndexType>
        <<<BlocksNum4ThreadsNum(params.count), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params);
  }

}

class PermuteImpl : public Permute, public CudaGraphSupport {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteImpl);
  PermuteImpl() = default;
  ~PermuteImpl() override = default;

  using Permute::Launch;
  void Launch(StreamContext* stream_ctx, DataType data_type, size_t num_dims,
              const int64_t* src_dims, const void* src, const int* permutation,
              void* dst) override {
    SimplifyThenLaunch(stream_ctx, data_type, num_dims, src_dims, src, permutation, dst);
  }
};

class PermuteFactoryImpl : public PermuteFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteFactoryImpl);
  PermuteFactoryImpl() = default;
  ~PermuteFactoryImpl() override = default;

  std::unique_ptr<Permute> New(size_t max_num_dims) override {
    if (max_num_dims <= kMaxNumDims) {
      return std::unique_ptr<Permute>(new PermuteImpl());
    } else {
      return nullptr;
    }
  }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kGPU, PermuteFactory, PermuteFactoryImpl);

}  // namespace

}  // namespace permute_internal

}  // namespace primitive

}  // namespace oneflow
