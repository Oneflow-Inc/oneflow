#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/common/nd_index_offset_helper.h"
#include "oneflow/core/primitive/include/permute.h"
#include "oneflow/core/primitive/common/permute.h"
#include "oneflow/core/stream/cuda_stream_context.h"
#include "oneflow/core/primitive/cuda/cuda_graph_support.h"
#include <hip/hip_runtime.h>

namespace oneflow {

namespace primitive {

namespace permute_internal {

namespace {

constexpr int32_t TILE_SIZE = 32; 
constexpr int32_t kBlockRows = 8; 

template<size_t num_dims, size_t movement_size, typename IndexType>
__global__ void PermuteKernel(PermuteKernelParams<num_dims, IndexType> params) {
  using T = typename std::aligned_storage<movement_size, movement_size>::type;
  const T* src = reinterpret_cast<const T*>(params.src);
  T* dst = reinterpret_cast<T*>(params.dst);
  CUDA_1D_KERNEL_LOOP_T(IndexType, i, params.count) {
    IndexType src_index[num_dims];
    IndexType dst_index[num_dims];
    params.dst_index_helper.OffsetToNdIndex(i, dst_index);
#pragma unroll
    for (size_t dim = 0; dim < num_dims; ++dim) {
      src_index[params.permutation[dim]] = dst_index[dim];
    }
    IndexType src_offset = params.src_index_helper.NdIndexToOffset(src_index);
    dst[i] = src[src_offset];
  }
}

// (B, X, Y) -> (B, Y, X), refer from https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
template<size_t movement_size, typename IndexType>
__global__ void BatchPermuteKernel(PermuteKernelParams<3, IndexType> params, 
                                   IndexType N, 
                                   IndexType H, 
                                   IndexType W, 
                                   IndexType dh, 
                                   IndexType dw) {
  using T = typename std::aligned_storage<movement_size, movement_size>::type;
  __shared__ T tile[TILE_SIZE][TILE_SIZE+1]; // To avoid bank conflict. 
  const T* src = reinterpret_cast<const T*>(params.src);
  T* dst = reinterpret_cast<T*>(params.dst);
  const IndexType n = blockIdx.x / (dh * dw); // the index of batch. 
  const IndexType k = blockIdx.x % (dh * dw); // the flatten index of tile in a batch. 
  const IndexType r = k / dw; // the row index of tile in a batch. 
  const IndexType c = k % dw; // the col index of tile in a batch. 
  const IndexType offset = n * H * W; 
  int x = c * TILE_SIZE + threadIdx.x; 
  int y = r * TILE_SIZE + threadIdx.y;
  if (x < W) {
    #pragma unroll 
    // each thread process 4 elements. 
    for (int i = 0; threadIdx.y + i < TILE_SIZE && y + i < H; i += kBlockRows) {
      tile[threadIdx.y + i][threadIdx.x] = src[offset + (y + i) * W + x];
      }
    }
  __syncthreads();
  x = r * TILE_SIZE + threadIdx.x;
  y = c * TILE_SIZE + threadIdx.y;
  if (x < H) {
    #pragma unroll 
    for (int i = 0; threadIdx.y + i < TILE_SIZE && y + i < W; i += kBlockRows) {
      dst[offset + (y + i) * H + x] = tile[threadIdx.x][threadIdx.y + i];
    }
  }
}

template<size_t num_dims, size_t movement_size, typename IndexType>
void LaunchBatchPermuteKernel(StreamContext* stream_ctx, PermuteKernelParams<num_dims, IndexType> params) {
  hipStream_t hip_stream =
      CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();

  IndexType global_index[3]; 
  params.src_index_helper.OffsetToNdIndex(params.count, global_index); 
  const IndexType N = global_index[0]; 
  const IndexType H = global_index[1]; 
  const IndexType W = global_index[2]; 

  const IndexType dh = H / TILE_SIZE; 
  const IndexType dw = W / TILE_SIZE; 
  int32_t batch_permute_block_size = 1024; // 32 * 32 == share memory tile size. 
  int32_t grid_size = std::min((params.count + batch_permute_block_size - 1) / batch_permute_block_size, kCudaMaxBlocksNum);
  BatchPermuteKernel<num_dims, movement_size, IndexType>
      <<<grid_size, dim3(32, 8), 0, hip_stream>>>(params); // Set threads num as 32x8 cause each threads process 4 elements to 32x32 share memory. 
}

template<size_t num_dims, typename IndexType>
bool CheckLaunchBatchPermute(PermuteKernelParams<num_dims, IndexType> params){
  // (0, 1, 2) -> (0, 2, 1)
  if(num_dims==3){
    if(params.permutation[num_dims-1]==num_dims-2 && params.permutation[num_dims-2]==num_dims-1){
    return true; 
   }
  }
  return false; 
}

template<size_t num_dims, size_t movement_size, typename IndexType>
void LaunchKernel(StreamContext* stream_ctx, PermuteKernelParams<num_dims, IndexType> params) {
  hipStream_t hip_stream =
      CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();
  if(CheckLaunchBatchPermute(params)){
    LaunchBatchPermuteKernel(stream_ctx, params); 
  }else{
    PermuteKernel<num_dims, movement_size, IndexType>
      <<<BlocksNum4ThreadsNum(params.count), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params);
  }
}

class PermuteImpl : public Permute, public CudaGraphSupport {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteImpl);
  PermuteImpl() = default;
  ~PermuteImpl() override = default;

  void Launch(StreamContext* stream_ctx, DataType data_type, size_t num_dims,
              const int64_t* src_dims, const void* src, const int* permutation,
              void* dst) override {
    SimplifyThenLaunch(stream_ctx, data_type, num_dims, src_dims, src, permutation, dst);
  }
};

class PermuteFactoryImpl : public PermuteFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteFactoryImpl);
  PermuteFactoryImpl() = default;
  ~PermuteFactoryImpl() override = default;

  std::unique_ptr<Permute> New(size_t max_num_dims) override {
    if (max_num_dims <= kMaxNumDims) {
      return std::unique_ptr<Permute>(new PermuteImpl());
    } else {
      return nullptr;
    }
  }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kGPU, PermuteFactory, PermuteFactoryImpl);

}  // namespace

}  // namespace permute_internal

}  // namespace primitive

}  // namespace oneflow
