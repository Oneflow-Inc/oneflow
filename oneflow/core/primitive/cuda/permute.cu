#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/primitive/include/permute.h"
#include "oneflow/core/primitive/common/permute_impl.h"
#include "oneflow/core/stream/cuda/cuda_stream_context.h"
#include <hip/hip_runtime.h>

namespace oneflow {

namespace primitive {

namespace permute {

namespace internal {

namespace {

constexpr int32_t kMov4TileSize = 32;
constexpr int32_t kMov2TileSize = 64;
constexpr int32_t kBlockRows = 8;

template<size_t num_dims, size_t movement_size, typename IndexType>
__global__ void PermuteKernel(PermuteKernelParams<num_dims, IndexType> params) {
  using T = typename std::aligned_storage<movement_size, movement_size>::type;
  const T* src = reinterpret_cast<const T*>(params.src);
  T* dst = reinterpret_cast<T*>(params.dst);
  IndexType src_index[num_dims];
  IndexType dst_index[num_dims];
  CUDA_1D_KERNEL_LOOP_T(IndexType, i, params.count) {
    params.dst_index_helper.OffsetToNdIndex(i, dst_index);
#pragma unroll
    for (size_t dim = 0; dim < num_dims; ++dim) {
      src_index[params.permutation[dim]] = dst_index[dim];
    }
    IndexType src_offset = params.src_index_helper.NdIndexToOffset(src_index);
    dst[i] = src[src_offset];
  }
}

// (B, X, Y) -> (B, Y, X)
// refer from https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
template<size_t num_dims, size_t movement_size, size_t tile_size, typename IndexType>
__global__ void BatchTransposeKernel(const void* src_ptr, void* dst_ptr, IndexType H, IndexType W,
                                     IndexType num_tile_rows, IndexType num_tile_cols,
                                     int32_t block_nums) {
  using T = typename std::aligned_storage<movement_size, movement_size>::type;
  __shared__ T tile[tile_size][tile_size + 1];  // To avoid bank conflict.

  const T* src = reinterpret_cast<const T*>(src_ptr);
  T* dst = reinterpret_cast<T*>(dst_ptr);

  IndexType batch_num_tile = num_tile_rows * num_tile_cols;
  for (int i = blockIdx.x, step = gridDim.x; i < block_nums; i += step) {
    const IndexType batch_index = i / batch_num_tile;  // the index of batch.
    const IndexType flatten_index =
        i - batch_index * batch_num_tile;  // equal to i % (num_tile_rows*num_tile_cols). the
                                           // flatten index of tile in a batch.

    const IndexType row_index = flatten_index / num_tile_cols;  // the row index of tile in a batch.
    const IndexType col_index =
        flatten_index
        - row_index
              * num_tile_cols;  // equal to k % num_tile_cols. the col index of tile in a batch.
    const IndexType offset = batch_index * H * W;
    IndexType x = col_index * tile_size + threadIdx.x;
    IndexType y = row_index * tile_size + threadIdx.y;
    if (x < W) {
      IndexType y_range =
          ((tile_size - threadIdx.y) < (H - y)) ? (tile_size - threadIdx.y) : (H - y);
#pragma unroll
      // each thread process 4 elements.
      // `i < y_range` equals to: `threadIdx.y + i < tile_size && y + i < H`.
      for (int i = 0; i < y_range; i += kBlockRows) {
        tile[threadIdx.y + i][threadIdx.x] = src[offset + (y + i) * W + x];
      }
    }
    __syncthreads();
    x = row_index * tile_size + threadIdx.x;
    y = col_index * tile_size + threadIdx.y;
    if (x < H) {
      IndexType x_range =
          ((tile_size - threadIdx.y) < (W - y)) ? (tile_size - threadIdx.y) : (W - y);
#pragma unroll
      // `i < x_range` equals to: `threadIdx.y + i < tile_size && y + i < W`.
      for (int i = 0; i < x_range; i += kBlockRows) {
        dst[offset + (y + i) * H + x] = tile[threadIdx.x][threadIdx.y + i];
      }
    }
    __syncthreads();
  }
}

/*
Here is a Movementsie=2 version of Batch Transpose.
When the H W can be divided by 2. we can read data use movementsize=4, and write back as
movementsize=2.
*/
template<size_t num_dims, size_t tile_size, typename IndexType>
__global__ void BatchTransposeMovement2Kernel(const void* src_ptr, void* dst_ptr, IndexType rows,
                                              IndexType cols, IndexType num_tile_rows,
                                              IndexType num_tile_cols, int32_t block_nums) {
  static_assert(tile_size % 2 == 0, "");
  using T_MOV2 = typename std::aligned_storage<2, 2>::type;
  using T_MOV4 = typename std::aligned_storage<4, 4>::type;

  const T_MOV4* src = reinterpret_cast<const T_MOV4*>(src_ptr);
  T_MOV4* dst = reinterpret_cast<T_MOV4*>(dst_ptr);

  // Use union structure to process Load and Store.
  __shared__ union {
    T_MOV2 tile_m2[tile_size][tile_size + 2];      // half [64][66]
    T_MOV4 tile_m4[tile_size][tile_size / 2 + 1];  // half2 [64][33]
  } tile_mem;

  IndexType batch_num_tile = num_tile_rows * num_tile_cols;
  for (int i = blockIdx.x, step = gridDim.x; i < block_nums; i += step) {
    const IndexType batch_index = i / batch_num_tile;  // the index of batch.
    const IndexType flatten_index =
        i - batch_index * batch_num_tile;  // equal to i%(num_tile_rows*num_tile_cols). the flatten
                                           // index of tile in a batch.

    const IndexType row_index = flatten_index / num_tile_cols;  // the row index of tile in a batch.
    const IndexType col_index =
        flatten_index
        - row_index
              * num_tile_cols;  // equal to k % num_tile_cols. the col index of tile in a batch.
    const IndexType offset = batch_index * rows * cols;
    IndexType x =
        col_index * tile_size + threadIdx.x * 2;  // cause each thread process a half2 element, we
                                                  // need to multiply 2 for threadIdx.x.
    IndexType y = row_index * tile_size + threadIdx.y;
    if (x < cols) {
      // each thread process 4 elements.
      IndexType y_range =
          ((tile_size - threadIdx.y) < (rows - y)) ? (tile_size - threadIdx.y) : (rows - y);
#pragma unroll
      // `i < y_range` equals to: `threadIdx.y + i < tile_size && y + i < rows`.
      for (int i = 0; i < y_range; i += kBlockRows) {
        // each thread load a half2.
        tile_mem.tile_m4[threadIdx.y + i][threadIdx.x] = src[(offset + (y + i) * cols + x) / 2];
      }
    }
    __syncthreads();
    x = row_index * tile_size + threadIdx.x * 2;  // cause each thread process a half2 element, we
                                                  // need to multiply 2 for threadIdx.x.
    y = col_index * tile_size + threadIdx.y;
    if (x < rows) {
      IndexType x_range =
          ((tile_size - threadIdx.y) < (cols - y)) ? (tile_size - threadIdx.y) : (cols - y);
#pragma unroll
      // `i < x_range` equals to: `threadIdx.y + i < tile_size && y + i < cols`.
      for (int i = 0; i < x_range; i += kBlockRows) {
        /*
        When write back as column, it cannot be stored as half2 directly.
        So we split as 2 half elements, and write back separately.
        */
        union {
          T_MOV4 m4;
          T_MOV2 m2[2];
        } tmp_storage;
        tmp_storage.m2[0] = tile_mem.tile_m2[threadIdx.x * 2][threadIdx.y + i];
        tmp_storage.m2[1] = tile_mem.tile_m2[threadIdx.x * 2 + 1][threadIdx.y + i];
        dst[(offset + (y + i) * rows + x) / 2] = tmp_storage.m4;
      }
    }
    __syncthreads();
  }
}

template<size_t num_dims, size_t movement_size, size_t tile_size, typename IndexType>
void LaunchBatchTransposeKernel(hipStream_t& hip_stream,
                                const PermuteKernelParams<num_dims, IndexType>& params,
                                const IndexType& num_batches, const IndexType& rows,
                                const IndexType& cols) {
  IndexType num_tile_rows = (rows + tile_size - 1) / tile_size;
  IndexType num_tile_cols = (cols + tile_size - 1) / tile_size;

  const int32_t block_nums = num_batches * num_tile_rows * num_tile_cols;
  int32_t checked_block_nums = std::min(block_nums, kCudaMaxBlocksNum);
  if (tile_size == kMov2TileSize) {
    const int32_t half2_thread = tile_size / 2;  // cause each thread process two half elements.
    BatchTransposeMovement2Kernel<num_dims, kMov2TileSize, IndexType>
        <<<checked_block_nums, dim3(half2_thread, kBlockRows), 0, hip_stream>>>(
            params.src, params.dst, rows, cols, num_tile_rows, num_tile_cols,
            block_nums);  // Set threads num as 32x8 cause each threads
                          // process 4 elements to 32x32 share memory.
  } else {
    BatchTransposeKernel<num_dims, movement_size, tile_size, IndexType>
        <<<checked_block_nums, dim3(tile_size, kBlockRows), 0, hip_stream>>>(
            params.src, params.dst, rows, cols, num_tile_rows, num_tile_cols, block_nums);
  }
}

template<size_t tile_size, typename IndexType>
bool CheckIfGreaterEqualThanTileSize(const IndexType& rows, const IndexType& cols) {
  if (rows < tile_size || cols < tile_size) { return false; }
  return true;
}

template<size_t num_dims, size_t tile_size, typename IndexType>
bool CheckLaunchBatchTranspose(const int* permutation, const IndexType& num_batches,
                               const IndexType& rows, const IndexType& cols) {
  if (CheckIfGreaterEqualThanTileSize<tile_size, IndexType>(rows, cols)) {
    if (num_batches == 1) {
      // 2d tensor case: (0, 1) -> (1, 0)
      return true;
    } else if (num_dims == 3 && permutation[2] == 1 && permutation[1] == 2) {
      // 3d tensor case: (0, 1, 2) -> (0, 2, 1)
      return true;
    } else {
      return false;
    }
  }
  return false;
}

template<typename IndexType, size_t movement_size>
bool CheckUseMov2(const IndexType& rows, const IndexType& cols, const void* src, void* dst) {
  auto src_ptr = reinterpret_cast<std::uintptr_t>(src);
  auto dst_ptr = reinterpret_cast<std::uintptr_t>(dst);
  return (movement_size == 2) && (rows % 2 == 0) && (cols % 2 == 0) && (src_ptr % 4 == 0)
         && (dst_ptr % 4 == 0);
  ;
}

template<size_t num_dims, typename IndexType>
void InferBatchTransposeShape(const int64_t* src_dims, IndexType* num_batches, IndexType* rows,
                              IndexType* cols) {
  if (num_dims == 2) {
    *num_batches = 1;
    *rows = src_dims[0];
    *cols = src_dims[1];
  } else {
    *num_batches = src_dims[0];
    *rows = src_dims[1];
    *cols = src_dims[2];
  }
}

template<size_t num_dims, size_t movement_size, typename IndexType>
void LaunchKernel(StreamContext* stream_ctx, const int64_t* src_dims, const void* src,
                  const int* permutation, void* dst, size_t count) {
  PermuteKernelParams<num_dims, IndexType> params =
      MakePermuteParams<num_dims, IndexType>(src_dims, src, permutation, dst, count);
  hipStream_t hip_stream =
      CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();

  if (num_dims == 2 || num_dims == 3) {
    IndexType num_batches;
    IndexType rows;
    IndexType cols;
    InferBatchTransposeShape<num_dims, IndexType>(src_dims, &num_batches, &rows, &cols);
    if (CheckLaunchBatchTranspose<num_dims, kMov4TileSize>(params.permutation, num_batches, rows,
                                                           cols)) {
      if (CheckUseMov2<IndexType, movement_size>(rows, cols, src, dst)) {
        LaunchBatchTransposeKernel<num_dims, 2, kMov2TileSize, IndexType>(hip_stream, params,
                                                                          num_batches, rows, cols);
      } else {
        LaunchBatchTransposeKernel<num_dims, movement_size, kMov4TileSize, IndexType>(
            hip_stream, params, num_batches, rows, cols);
      }
    } else {
      PermuteKernel<num_dims, movement_size, IndexType>
          <<<BlocksNum4ThreadsNum(params.count), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params);
    }
  } else {
    PermuteKernel<num_dims, movement_size, IndexType>
        <<<BlocksNum4ThreadsNum(params.count), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params);
  }
}

class PermuteImpl : public Permute {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteImpl);
  PermuteImpl() = default;
  ~PermuteImpl() override = default;

  using Permute::Launch;
  void Launch(StreamContext* stream_ctx, DataType data_type, size_t num_dims,
              const int64_t* src_dims, const void* src, const int* permutation,
              void* dst) override {
    SimplifyThenLaunch(stream_ctx, data_type, num_dims, src_dims, src, permutation, dst);
  }
};

class PermuteFactoryImpl : public PermuteFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteFactoryImpl);
  PermuteFactoryImpl() = default;
  ~PermuteFactoryImpl() override = default;

  std::unique_ptr<Permute> New(size_t max_num_dims) override {
    if (max_num_dims <= kMaxNumDims) {
      return std::unique_ptr<Permute>(new PermuteImpl());
    } else {
      return nullptr;
    }
  }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kGPU, PermuteFactory, PermuteFactoryImpl);

}  // namespace

}  // namespace internal

}  // namespace permute

}  // namespace primitive

}  // namespace oneflow
