#include "oneflow/core/blas/kernel_util.h"
#include "oneflow/core/actor/cuda_device_context.h"

namespace oneflow {

template<>
void KernelUtil<DeviceType::kGPU, float>::Memcpy(
    const KernelCtx& ctx, void* dst, const void* src, size_t sz) {
  CHECK_EQ(hipMemcpyAsync(dst, src, sz, hipMemcpyDeviceToDevice, 
      ctx.device_ctx->hip_stream()), hipSuccess);
}

template<>
void KernelUtil<DeviceType::kGPU, double>::Memcpy(
    const KernelCtx& ctx, void* dst, const void* src, size_t sz) {
  CHECK_EQ(hipMemcpyAsync(dst, src, sz, hipMemcpyDeviceToDevice, 
      ctx.device_ctx->hip_stream()), hipSuccess);
}

template<>
void KernelUtil<DeviceType::kGPU, float>::Memset(
    const KernelCtx& ctx, void* dst, const char value, size_t sz) {
  CHECK_EQ(hipMemsetAsync(dst, value, sz, ctx.device_ctx->hip_stream()), 
      hipSuccess);
}

template<>
void KernelUtil<DeviceType::kGPU, double>::Memset(
    const KernelCtx& ctx, void* dst, const char value, size_t sz) {
  CHECK_EQ(hipMemsetAsync(dst, value, sz, ctx.device_ctx->hip_stream()), 
      hipSuccess);
}

template<>
void KernelUtil<DeviceType::kGPU, float>::BlasAxpy(
    const KernelCtx& ctx, const int N, const float alpha, const float *X, 
    const int incX, float *Y, const int incY) {
  float tmp_alpha = alpha;
  CHECK_EQ(hipblasSaxpy(
        ctx.device_ctx->cublas_handle(),
        N, &tmp_alpha, X, incX, Y, incY), 
      hipSuccess);
}

template<>
void KernelUtil<DeviceType::kGPU, double>::BlasAxpy(
    const KernelCtx& ctx, const int N, const double alpha, const double *X, 
    const int incX, double *Y, const int incY) {
  double tmp_alpha = alpha;
  CHECK_EQ(hipblasDaxpy(
        ctx.device_ctx->cublas_handle(),
        N, &tmp_alpha, X, incX, Y, incY), 
      hipSuccess);
}

template<> 
void KernelUtil<DeviceType::kGPU, float>::BlasScal(
    const KernelCtx& ctx, const int n, const float alpha, float* x, int incx) {
  float tmp_alpha = alpha;
  CHECK_EQ(hipblasSscal(
        ctx.device_ctx->cublas_handle(),
        n, &tmp_alpha, x, incx), 
      hipSuccess);
}

template<> 
void KernelUtil<DeviceType::kGPU, double>::BlasScal(
    const KernelCtx& ctx, const int n, const double alpha, double* x,
    int incx) {
  double tmp_alpha = alpha;
  CHECK_EQ(hipblasDscal(
        ctx.device_ctx->cublas_handle(),
        n, &tmp_alpha, x, incx), 
      hipSuccess);
}

}  // namespace oneflow
