#include "oneflow/core/blas/cublas_template.h"

namespace oneflow {

// level 1 vector and vector
// dot product
template<>
void cublas_dot<float>(hipblasHandle_t handle, int n, const float* x, int incx,
                       const float* y, int incy, float* result) {
  CudaCheck(hipblasSdot(handle, n, x, incx, y, incy, result));
}

template<>
void cublas_dot<double>(hipblasHandle_t handle, int n, const double* x, int incx,
                        const double* y, int incy, double* result) {
  CudaCheck(hipblasDdot(handle, n, x, incx, y, incy, result));
}

// swap x and y
template<>
void cublas_swap<float>(hipblasHandle_t handle, int n, float* x, int incx,
                        float* y, int incy) {
  CudaCheck(hipblasSswap(handle, n, x, incx, y, incy));
}

template<>
void cublas_swap<double>(hipblasHandle_t handle, int n, double* x, int incx,
                         double* y, int incy) {
  CudaCheck(hipblasDswap(handle, n, x, incx, y, incy));
}

// copy x into y
template<>
void cublas_copy<float>(hipblasHandle_t handle, int n, const float* x, int incx,
                        float* y, int incy) {
  CudaCheck(hipblasScopy(handle, n, x, incx, y, incy));
}

template<>
void cublas_copy<double>(hipblasHandle_t handle, int n, const double* x,
                         int incx, double* y, int incy) {
  CudaCheck(hipblasDcopy(handle, n, x, incx, y, incy));
}

// y = a*x + y
template<>
void cublas_axpy<float>(hipblasHandle_t handle, int n, const float* alpha,
                        const float* x, const int incx, float* y,
                        const int incy) {
  CudaCheck(hipblasSaxpy(handle, n, alpha, x, incx, y, incy));
}

template<>
void cublas_axpy<double>(hipblasHandle_t handle, int n, const double* alpha,
                         const double* x, const int incx, double* y, int incy) {
  CudaCheck(hipblasDaxpy(handle, n, alpha, x, incx, y, incy));
}

// x = a*x
template<>
void cublas_scal<float>(hipblasHandle_t handle, int n, const float* alpha,
                        float* x, int incx) {
  CudaCheck(hipblasSscal(handle, n, alpha, x, incx));
}

template<>
void cublas_scal<double>(hipblasHandle_t handle, int n, const double* alpha,
                         double* x, int incx) {
  CudaCheck(hipblasDscal(handle, n, alpha, x, incx));
}

// level 2 matrix and vector
// matrix vector multiply
template<>
void cublas_gemv<float>(hipblasHandle_t handle, hipblasOperation_t trans, int m,
                        int n, const float* alpha, const float* a, int lda,
                        const float* x, int incx, const float* beta, float* y,
                        int incy) {
  CudaCheck(
      hipblasSgemv(handle, trans, m, n, alpha, a, lda, x, incx, beta, y, incy));
}

template<>
void cublas_gemv<double>(hipblasHandle_t handle, hipblasOperation_t trans, int m,
                         int n, const double* alpha, const double* a, int lda,
                         const double* x, int incx, const double* beta,
                         double* y, int incy) {
  CudaCheck(
      hipblasDgemv(handle, trans, m, n, alpha, a, lda, x, incx, beta, y, incy));
}

// level 3 matrix and matrix
// matrix matrix multiply
template<>
void cublas_gemm<float>(hipblasHandle_t handle, hipblasOperation_t cutrans_a,
                        hipblasOperation_t cutrans_b, int m, int n, int k,
                        const float* alpha, const float* a, int lda,
                        const float* b, int ldb, const float* beta, float* c,
                        int ldc) {
  CudaCheck(hipblasSgemm(handle, cutrans_a, cutrans_b, m, n, k, alpha, a, lda, b,
                        ldb, beta, c, ldc));
}

template<>
void cublas_gemm<double>(hipblasHandle_t handle, hipblasOperation_t cutrans_a,
                         hipblasOperation_t cutrans_b, int m, int n, int k,
                         const double* alpha, const double* a, int lda,
                         const double* b, int ldb, const double* beta,
                         double* c, int ldc) {
  CudaCheck(hipblasDgemm(handle, cutrans_a, cutrans_b, m, n, k, alpha, a, lda, b,
                        ldb, beta, c, ldc));
}

}  // namespace oneflow
