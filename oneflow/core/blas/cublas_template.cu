#include "oneflow/core/blas/cublas_template.h"

namespace oneflow {

// level 1 vector and vector
// dot product
template<>
void cublas_dot<float>(
    hipblasHandle_t handle, int n, const float* x, int incx, const float* y,
    int incy, float* result) {
  CHECK_EQ(hipblasSdot(handle, n, x, incx, y, incy, result),
           HIPBLAS_STATUS_SUCCESS);
}

template<>
void cublas_dot<double>(
    hipblasHandle_t handle, int n, const double* x, int incx, const double* y,
    int incy, double* result) {
  CHECK_EQ(hipblasDdot(handle, n, x, incx, y, incy, result),
           HIPBLAS_STATUS_SUCCESS);
}

// swap x and y
template<>
void cublas_swap<float>(
    hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy) {
  CHECK_EQ(hipblasSswap(handle, n, x, incx, y, incy), HIPBLAS_STATUS_SUCCESS);
}

template<>
void cublas_swap<double>(
    hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy) {
  CHECK_EQ(hipblasDswap(handle, n, x, incx, y, incy), HIPBLAS_STATUS_SUCCESS);
}

// copy x into y
template<>
void cublas_copy<float>(
    hipblasHandle_t handle, int n, const float* x, int incx,
    float* y, int incy) {
  CHECK_EQ(hipblasScopy(handle, n, x, incx, y, incy), HIPBLAS_STATUS_SUCCESS);
}

template<>
void cublas_copy<double>(
    hipblasHandle_t handle, int n, const double* x, int incx,
    double* y, int incy) {
  CHECK_EQ(hipblasDcopy(handle, n, x, incx, y, incy), HIPBLAS_STATUS_SUCCESS);
}

// y = a*x + y
template<>
void cublas_axpy<float>(
    hipblasHandle_t handle, int n, const float* alpha, const float* x, 
    const int incx, float* y, const int incy) {
  CHECK_EQ(hipblasSaxpy(handle, n, alpha, x, incx, y, incy),
           HIPBLAS_STATUS_SUCCESS);
}

template<>
void cublas_axpy<double>(
    hipblasHandle_t handle, int n, const double* alpha, 
    const double* x, const int incx,
    double* y, int incy) {
  CHECK_EQ(hipblasDaxpy(handle, n, alpha, x, incx, y, incy),
           HIPBLAS_STATUS_SUCCESS);
}

// x = a*x
template<>
void cublas_scal<float>(
    hipblasHandle_t handle, int n, const float* alpha, float* x, int incx) {
  CHECK_EQ(hipblasSscal(handle, n, alpha, x, incx), HIPBLAS_STATUS_SUCCESS);
}

template<>
void cublas_scal<double>(
    hipblasHandle_t handle, int n, const double* alpha, double* x, int incx) {
  CHECK_EQ(hipblasDscal(handle, n, alpha, x, incx), HIPBLAS_STATUS_SUCCESS);
}

// level 2 matrix and vector
// matrix vector multiply
template<>
void cublas_gemv<float>(
    hipblasHandle_t handle, hipblasOperation_t trans, int m, int n,
    const float* alpha, const float* a, int lda, const float* x, int incx,
    const float* beta, float* y, int incy) {
  CHECK_EQ(hipblasSgemv(
               handle, trans, m, n, alpha, a, lda, x, incx, beta, y, incy),
           HIPBLAS_STATUS_SUCCESS);
}

template<>
void cublas_gemv<double>(
    hipblasHandle_t handle, hipblasOperation_t trans, int m, int n,
    const double* alpha, const double* a, int lda, const double* x, int incx,
    const double* beta, double* y, int incy) {
  CHECK_EQ(hipblasDgemv(
               handle, trans, m, n, alpha, a, lda, x, incx, beta, y, incy),
           HIPBLAS_STATUS_SUCCESS);
}

// level 3 matrix and matrix
// matrix matrix multiply
template<>
void cublas_gemm<float>(
    hipblasHandle_t handle, hipblasOperation_t cutrans_a,
    hipblasOperation_t cutrans_b, int m, int n, int k,
    const float* alpha, const float* a, int lda,
    const float* b, int ldb, const float* beta, float* c, int ldc) {
  CHECK_EQ(hipblasSgemm(
               handle, cutrans_a, cutrans_b, m, n, k, alpha, a, lda, b, ldb, beta,
               c, ldc),
           HIPBLAS_STATUS_SUCCESS);
}

template<>
void cublas_gemm<double>(
    hipblasHandle_t handle, hipblasOperation_t cutrans_a,
    hipblasOperation_t cutrans_b, int m, int n, int k,
    const double* alpha, const double* a, int lda,
    const double* b, int ldb, const double* beta, double* c, int ldc) {
  CHECK_EQ(hipblasDgemm(
               handle, cutrans_a, cutrans_b, m, n, k, alpha, a, lda, b, ldb, beta,
               c, ldc),
           HIPBLAS_STATUS_SUCCESS);
}

}  // namespace oneflow
