#include "oneflow/core/blas/cublas_template.h"

namespace oneflow {

template<>
void cublas_gemm<float>(
    const hipblasHandle_t& cublas_handle, hipblasOperation_t cuTransA,
    hipblasOperation_t cuTransB, int M, int N, int K,
    const float alpha, const float* A, int lda, const float* B,
    int ldb, const float beta, float* C, int ldc) {
  CHECK_EQ(hipblasSgemm(
               cublas_handle, cuTransA, cuTransB, M, N, K, &alpha, A, lda, B,
               ldb, &beta, C, ldc),
           HIPBLAS_STATUS_SUCCESS);
}

template<>
void cublas_gemm<double>(
    const hipblasHandle_t& cublas_handle, hipblasOperation_t cuTransA,
    hipblasOperation_t cuTransB, int M, int N, int K,
    const double alpha, const double* A, int lda, const double* B,
    int ldb, const double beta, double* C, int ldc) {
  CHECK_EQ(hipblasDgemm(
               cublas_handle, cuTransA, cuTransB, M, N, K, &alpha, A, lda, B,
               ldb, &beta, C, ldc),
           HIPBLAS_STATUS_SUCCESS);
}

template<>
void cublas_axpy<float>(
    hipblasHandle_t handle, int n,
    const float *alpha,
    const float *x, int incx,
    float *y, int incy) {
  CHECK_EQ(hipblasSaxpy(handle, n, alpha, x, incx, y, incy),
           HIPBLAS_STATUS_SUCCESS);
}

template<>
void cublas_axpy<double>(
    hipblasHandle_t handle, int n,
    const double *alpha,
    const double *x, int incx,
    double *y, int incy) {
  CHECK_EQ(hipblasDaxpy(handle, n, alpha, x, incx, y, incy),
          HIPBLAS_STATUS_SUCCESS);
}

}  // namespace oneflow
