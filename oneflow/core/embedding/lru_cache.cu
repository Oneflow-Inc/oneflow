#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/embedding/lru_cache.h"
#include "oneflow/core/device/cuda_util.h"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/embedding/hash_functions.cuh"
#include <cuda/std/semaphore>
#include <new>

namespace oneflow {

namespace embedding {

namespace {

constexpr int kWarpSize = 32;
constexpr int kNumWarpPerBlock = 4;
constexpr int kBlockSize = kNumWarpPerBlock * kWarpSize;
constexpr uint32_t kFullMask = 0xFFFFFFFFU;

ep::CudaLaunchConfig GetLaunchConfig(uint32_t n_keys) {
  return ep::CudaLaunchConfig((n_keys + kNumWarpPerBlock - 1) / kNumWarpPerBlock,
                              kWarpSize * kNumWarpPerBlock, 0);
}

template<typename Key, typename Elem>
struct LruCacheContext {
  Key* keys;
  Elem* lines;
  uint8_t* lru_queue;
  cuda::binary_semaphore<cuda::thread_scope_device>* mutex;
  uint64_t n_set;
  uint32_t line_size;
};

__global__ void InitCacheSetMutex(uint32_t n_set,
                                  cuda::binary_semaphore<cuda::thread_scope_device>* mutex) {
  const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n_set) { new (mutex + idx) cuda::binary_semaphore<cuda::thread_scope_device>(1); }
}

template<typename Key, typename Elem>
void ClearLruCacheContext(LruCacheContext<Key, Elem>* ctx) {
  OF_CUDA_CHECK(hipMemset(ctx->keys, 0, ctx->n_set * kWarpSize * sizeof(Key)));
  OF_CUDA_CHECK(hipMemset(ctx->lru_queue, 0, ctx->n_set * kWarpSize * sizeof(uint8_t)));
  InitCacheSetMutex<<<(ctx->n_set - 1 + 256) / 256, 256>>>(ctx->n_set, ctx->mutex);
}

template<typename Key, typename Elem>
void InitLruCacheContext(const CacheOptions& options, LruCacheContext<Key, Elem>* ctx) {
  const size_t key_size_per_set = kWarpSize * sizeof(Key);
  const uint32_t line_size = options.value_size / sizeof(Elem);
  const size_t lines_size_per_set = kWarpSize * line_size * sizeof(Elem);
  const size_t lru_size_per_set = kWarpSize * sizeof(uint8_t);
  const size_t mutex_size_per_set = sizeof(cuda::binary_semaphore<cuda::thread_scope_device>);
  const size_t size_per_set =
      key_size_per_set + lines_size_per_set + lru_size_per_set + mutex_size_per_set;
  const size_t n_set = (options.capacity - 1 + kWarpSize) / kWarpSize;
  CHECK_GT(n_set, 0);
  ctx->n_set = n_set;
  ctx->line_size = line_size;
  const size_t keys_size = n_set * key_size_per_set;
  OF_CUDA_CHECK(hipMalloc(&(ctx->keys), keys_size));
  const size_t lines_size = n_set * lines_size_per_set;
  OF_CUDA_CHECK(hipMalloc(&(ctx->lines), lines_size));
  const size_t lru_queue_size = n_set * lru_size_per_set;
  OF_CUDA_CHECK(hipMalloc(&(ctx->lru_queue), lru_queue_size));
  const size_t mutex_size = n_set * mutex_size_per_set;
  OF_CUDA_CHECK(hipMalloc(&(ctx->mutex), mutex_size));

  ClearLruCacheContext(ctx);
}

template<typename Key, typename Elem>
void DestroyLruCacheContext(LruCacheContext<Key, Elem>* ctx) {
  OF_CUDA_CHECK(hipFree(ctx->keys));
  OF_CUDA_CHECK(hipFree(ctx->lines));
  OF_CUDA_CHECK(hipFree(ctx->lru_queue));
  OF_CUDA_CHECK(hipFree(ctx->mutex));
}

struct ThreadContext {
  __device__ ThreadContext() {
    const uint32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    global_warp_id = global_thread_id / kWarpSize;
    warp_id_in_block = global_warp_id % kNumWarpPerBlock;
    num_warps = gridDim.x * kNumWarpPerBlock;
    lane_id = global_thread_id % kWarpSize;
  }

  uint32_t global_warp_id;
  uint32_t warp_id_in_block;
  uint32_t num_warps;
  uint32_t lane_id;
};

template<typename Key, typename Elem>
struct SetContext {
  __device__ SetContext(const LruCacheContext<Key, Elem>& ctx, uint32_t set_id) {
    keys = ctx.keys + set_id * kWarpSize;
    lines = ctx.lines + set_id * kWarpSize * ctx.line_size;
    lru_queue = ctx.lru_queue + set_id * kWarpSize;
    mutex = ctx.mutex + set_id;
  }

  __device__ int Lookup(const ThreadContext& thread_ctx, Key key) {
    const bool lane_hit = (keys[thread_ctx.lane_id] == key);
    const unsigned hit_mask = __ballot_sync(kFullMask, lane_hit);
    if (hit_mask != 0) {
      return __ffs(static_cast<int>(hit_mask)) - 1;
    } else {
      return -1;
    }
  }

  __device__ void Read(const LruCacheContext<Key, Elem>& cache_ctx, const ThreadContext& thread_ctx,
                       int way, Elem* line) {
    const Elem* from_line = lines + way * cache_ctx.line_size;
    for (int i = thread_ctx.lane_id; i < cache_ctx.line_size; i += kWarpSize) {
      line[i] = from_line[i];
    }
  }

  __device__ int InsertWithoutEvicting(const LruCacheContext<Key, Elem>& cache_ctx,
                                       const ThreadContext& thread_ctx, Key key) {
    int lru_way_idx = -1;
    int insert_way = -1;
    const Key lane_key = keys[thread_ctx.lane_id];
    const unsigned hit_mask = __ballot_sync(kFullMask, lane_key == key);
    if (hit_mask != 0) {
      lru_way_idx = lru_queue[thread_ctx.lane_id];
      insert_way = __ffs(static_cast<int>(hit_mask)) - 1;
      int lru_lane_id = __ffs(__ballot_sync(kFullMask, lru_way_idx == insert_way)) - 1;
      if (thread_ctx.lane_id <= lru_lane_id) {
        lru_way_idx = __shfl_up_sync(__activemask(), lru_way_idx, 1);
      }
      if (thread_ctx.lane_id == 0) { lru_way_idx = insert_way; }
      __syncwarp();
    }
    if (insert_way == -1) {
      const unsigned valid_mask = __ballot_sync(kFullMask, lane_key != 0);
      if (valid_mask != kFullMask) {
        lru_way_idx = lru_queue[thread_ctx.lane_id];
        insert_way = __popc(static_cast<int>(valid_mask));
        lru_way_idx = __shfl_up_sync(kFullMask, lru_way_idx, 1);
        if (thread_ctx.lane_id == 0) {
          lru_way_idx = insert_way;
          keys[insert_way] = key;
        }
        __syncwarp();
      }
    }
    if (lru_way_idx != -1) { lru_queue[thread_ctx.lane_id] = lru_way_idx; }
    return insert_way;
  }

  __device__ void Evict(const LruCacheContext<Key, Elem>& cache_ctx,
                        const ThreadContext& thread_ctx, Key key, int* way, Key* evicted_key) {
    int lru_way_idx = lru_queue[thread_ctx.lane_id];
    const Key lane_key = keys[thread_ctx.lane_id];
    int insert_way = -1;
    const unsigned hit_mask = __ballot_sync(kFullMask, lane_key == key);
    insert_way = __shfl_sync(kFullMask, lru_way_idx, kWarpSize - 1);
    lru_way_idx = __shfl_up_sync(kFullMask, lru_way_idx, 1);
    *evicted_key = __shfl_sync(kFullMask, lane_key, insert_way);
    if (thread_ctx.lane_id == 0) {
      lru_way_idx = insert_way;
      keys[insert_way] = key;
    }
    __syncwarp();
    lru_queue[thread_ctx.lane_id] = lru_way_idx;
    *way = insert_way;
  }

  __device__ void Write(const LruCacheContext<Key, Elem>& cache_ctx,
                        const ThreadContext& thread_ctx, int way, const Elem* line) {
    Elem* to_line = lines + way * cache_ctx.line_size;
    for (int i = thread_ctx.lane_id; i < cache_ctx.line_size; i += kWarpSize) {
      to_line[i] = line[i];
    }
  }

  __device__ void Lock(const ThreadContext& thread_ctx) {
    if (thread_ctx.lane_id == 0) { mutex->acquire(); }
    __syncwarp();
  }

  __device__ void Unlock(const ThreadContext& thread_ctx) {
    if (thread_ctx.lane_id == 0) { mutex->release(); }
    __syncwarp();
  }

  Key* keys;
  Elem* lines;
  uint8_t* lru_queue;
  cuda::binary_semaphore<cuda::thread_scope_device>* mutex;
};

template<typename Elem>
__device__ Elem Zero() {
  return 0;
}

template<>
__device__ ulonglong2 Zero<ulonglong2>() {
  return ulonglong2{0, 0};
}

template<typename Key, typename Elem, bool test_only>
__global__ void GetKernel(LruCacheContext<Key, Elem> cache_ctx, uint32_t num_keys, const Key* keys,
                          Elem* values, uint32_t* n_missing_keys, Key* missing_keys,
                          uint32_t* missing_indices) {
  ThreadContext thread_ctx{};
  __shared__ Key block_keys[kNumWarpPerBlock][kWarpSize];
  __shared__ size_t block_set_ids[kNumWarpPerBlock][kWarpSize];
  for (uint32_t batch_offset = thread_ctx.global_warp_id * kWarpSize; batch_offset < num_keys;
       batch_offset += thread_ctx.num_warps * kWarpSize) {
    const uint32_t n_batch_keys = min(kWarpSize, num_keys - batch_offset);
    if (thread_ctx.lane_id < n_batch_keys) {
      const Key key = keys[batch_offset + thread_ctx.lane_id];
      const size_t hash = XXH64()(key);
      const uint32_t set_id = hash % cache_ctx.n_set;
      block_keys[thread_ctx.warp_id_in_block][thread_ctx.lane_id] = key;
      block_set_ids[thread_ctx.warp_id_in_block][thread_ctx.lane_id] = set_id;
    }
    __syncwarp();
    uint32_t n_warp_missing = 0;
    Key warp_missing_key = 0;
    uint32_t warp_missing_index = 0;
    for (uint32_t i = 0; i < n_batch_keys; ++i) {
      const uint32_t key_idx = batch_offset + i;
      const Key key = block_keys[thread_ctx.warp_id_in_block][i];
      if (key == 0) {
        if (!test_only) {
          for (int j = thread_ctx.lane_id; j < cache_ctx.line_size; j += kWarpSize) {
            *(values + key_idx * cache_ctx.line_size + j) = Zero<Elem>();
          }
        }
        continue;
      }
      const size_t set_id = block_set_ids[thread_ctx.warp_id_in_block][i];
      SetContext<Key, Elem> set_ctx(cache_ctx, set_id);
      const int way = set_ctx.Lookup(thread_ctx, key);
      if (way < 0) {
        if (thread_ctx.lane_id == n_warp_missing) {
          warp_missing_key = key;
          warp_missing_index = key_idx;
        }
        __syncwarp();
        n_warp_missing += 1;
      } else if (!test_only) {
        set_ctx.Read(cache_ctx, thread_ctx, way, values + key_idx * cache_ctx.line_size);
      }
    }
    if (n_warp_missing > 0) {
      uint32_t base_missing_idx = 0;
      if (thread_ctx.lane_id == 0) { base_missing_idx = atomicAdd(n_missing_keys, n_warp_missing); }
      __syncwarp();
      base_missing_idx = __shfl_sync(kFullMask, base_missing_idx, 0);
      if (thread_ctx.lane_id < n_warp_missing) {
        missing_keys[base_missing_idx + thread_ctx.lane_id] = warp_missing_key;
        missing_indices[base_missing_idx + thread_ctx.lane_id] = warp_missing_index;
      }
      __syncwarp();
    }
    __syncwarp();
  }
}

template<typename Key, typename Elem>
__global__ void PutWithoutEvictingKernel(LruCacheContext<Key, Elem> cache_ctx, uint32_t num_keys,
                                         const Key* keys, const Elem* values, uint32_t* n_missing,
                                         Key* missing_keys, uint32_t* missing_indices) {
  ThreadContext thread_ctx{};
  __shared__ Key block_keys[kNumWarpPerBlock][kWarpSize];
  __shared__ size_t block_set_ids[kNumWarpPerBlock][kWarpSize];
  for (uint32_t batch_offset = thread_ctx.global_warp_id * kWarpSize; batch_offset < num_keys;
       batch_offset += thread_ctx.num_warps * kWarpSize) {
    const uint32_t n_batch_keys = min(kWarpSize, num_keys - batch_offset);
    if (thread_ctx.lane_id < n_batch_keys) {
      const Key key = keys[batch_offset + thread_ctx.lane_id];
      const size_t hash = XXH64()(key);
      const uint32_t set_id = hash % cache_ctx.n_set;
      block_keys[thread_ctx.warp_id_in_block][thread_ctx.lane_id] = key;
      block_set_ids[thread_ctx.warp_id_in_block][thread_ctx.lane_id] = set_id;
    }
    __syncwarp();
    uint32_t n_warp_missing = 0;
    Key warp_missing_key = 0;
    uint32_t warp_missing_index = 0;
    for (uint32_t i = 0; i < n_batch_keys; ++i) {
      const uint32_t key_idx = batch_offset + i;
      const Key key = block_keys[thread_ctx.warp_id_in_block][i];
      if (key == 0) { continue; }
      const size_t set_id = block_set_ids[thread_ctx.warp_id_in_block][i];
      SetContext<Key, Elem> set_ctx(cache_ctx, set_id);
      set_ctx.Lock(thread_ctx);
      Key evicted_key = 0;
      const int insert_way = set_ctx.InsertWithoutEvicting(cache_ctx, thread_ctx, key);
      if (insert_way >= 0) {
        set_ctx.Write(cache_ctx, thread_ctx, insert_way, values + cache_ctx.line_size * key_idx);
      } else {
        if (thread_ctx.lane_id == n_warp_missing) {
          warp_missing_key = key;
          warp_missing_index = key_idx;
        }
        __syncwarp();
        n_warp_missing += 1;
      }
      set_ctx.Unlock(thread_ctx);
    }
    if (n_warp_missing > 0) {
      uint32_t base_missing_idx = 0;
      if (thread_ctx.lane_id == 0) { base_missing_idx = atomicAdd(n_missing, n_warp_missing); }
      __syncwarp();
      base_missing_idx = __shfl_sync(kFullMask, base_missing_idx, 0);
      if (thread_ctx.lane_id < n_warp_missing) {
        missing_keys[base_missing_idx + thread_ctx.lane_id] = warp_missing_key;
        missing_indices[base_missing_idx + thread_ctx.lane_id] = warp_missing_index;
      }
      __syncwarp();
    }
  }
}

template<typename Key, typename Elem>
__global__ void EvictKernel(LruCacheContext<Key, Elem> cache_ctx, const Key* keys,
                            const uint32_t* indices, const Elem* values, const uint32_t* n_evict,
                            Key* evicted_keys, Elem* evicted_values) {
  ThreadContext thread_ctx{};
  uint32_t num_evict = *n_evict;
  __shared__ Key block_keys[kNumWarpPerBlock][kWarpSize];
  __shared__ size_t block_set_ids[kNumWarpPerBlock][kWarpSize];
  for (uint32_t batch_offset = thread_ctx.global_warp_id * kWarpSize; batch_offset < num_evict;
       batch_offset += thread_ctx.num_warps * kWarpSize) {
    const uint32_t n_batch_keys = min(kWarpSize, num_evict - batch_offset);
    if (thread_ctx.lane_id < n_batch_keys) {
      const Key key = keys[batch_offset + thread_ctx.lane_id];
      const size_t hash = XXH64()(key);
      const uint32_t set_id = hash % cache_ctx.n_set;
      block_keys[thread_ctx.warp_id_in_block][thread_ctx.lane_id] = key;
      block_set_ids[thread_ctx.warp_id_in_block][thread_ctx.lane_id] = set_id;
    }
    __syncwarp();
    for (uint32_t i = 0; i < n_batch_keys; ++i) {
      const uint32_t key_idx = batch_offset + i;
      const Key key = block_keys[thread_ctx.warp_id_in_block][i];
      const uint32_t set_id = block_set_ids[thread_ctx.warp_id_in_block][i];
      SetContext<Key, Elem> set_ctx(cache_ctx, set_id);
      set_ctx.Lock(thread_ctx);
      int evicted_way = -1;
      Key evicted_key = 0;
      set_ctx.Evict(cache_ctx, thread_ctx, key, &evicted_way, &evicted_key);
      if (thread_ctx.lane_id == 0) { evicted_keys[key_idx] = evicted_key; }
      __syncwarp();
      set_ctx.Read(cache_ctx, thread_ctx, evicted_way,
                   evicted_values + cache_ctx.line_size * key_idx);
      set_ctx.Write(cache_ctx, thread_ctx, evicted_way,
                    values + cache_ctx.line_size * indices[key_idx]);
      set_ctx.Unlock(thread_ctx);
    }
  }
}

template<typename Key, typename Elem>
__global__ void DumpKernel(LruCacheContext<Key, Elem> cache_ctx, size_t start_key_index,
                           size_t end_key_index, uint32_t* n_dumped, Key* keys, Elem* values) {
  ThreadContext thread_ctx{};
  __shared__ Key warp_keys[kNumWarpPerBlock][kWarpSize];
  for (uint32_t warp_start_key_index = start_key_index + thread_ctx.global_warp_id * kWarpSize;
       warp_start_key_index < end_key_index;
       warp_start_key_index += thread_ctx.num_warps * kWarpSize) {
    Key lane_key = 0;
    if (warp_start_key_index + thread_ctx.lane_id < end_key_index) {
      lane_key = cache_ctx.keys[warp_start_key_index + thread_ctx.lane_id];
    }
    __syncwarp();
    warp_keys[thread_ctx.warp_id_in_block][thread_ctx.lane_id] = lane_key;
    const int key_count = __popc(__ballot_sync(kFullMask, lane_key != 0));
    if (key_count == 0) { continue; }
    uint32_t offset = 0;
    if (thread_ctx.lane_id == 0) { offset = atomicAdd(n_dumped, key_count); }
    offset = __shfl_sync(kFullMask, offset, 0);
    __syncwarp();
    for (uint32_t i = 0; i < kWarpSize; ++i) {
      const Key key = warp_keys[thread_ctx.warp_id_in_block][i];
      if (key == 0) { continue; }
      if (thread_ctx.lane_id == 0) { keys[offset] = key; }
      __syncwarp();
      for (uint32_t j = thread_ctx.lane_id; j < cache_ctx.line_size; j += kWarpSize) {
        values[offset * cache_ctx.line_size + j] =
            cache_ctx.lines[(warp_start_key_index + i) * cache_ctx.line_size + j];
      }
      __syncwarp();
      offset += 1;
    }
  }
}

template<typename Key, typename Elem>
class LruCache : public Cache {
 public:
  OF_DISALLOW_COPY_AND_MOVE(LruCache);
  explicit LruCache(const CacheOptions& options) : device_index_{}, max_query_length_(0) {
    OF_CUDA_CHECK(hipGetDevice(&device_index_));
    InitLruCacheContext(options, &ctx_);
  }
  ~LruCache() override {
    CudaCurrentDeviceGuard guard(device_index_);
    if (max_query_length_ != 0) {
      OF_CUDA_CHECK(hipFree(query_indices_buffer_));
      OF_CUDA_CHECK(hipFree(query_keys_buffer_));
    }
    DestroyLruCacheContext(&ctx_);
  }

  uint32_t KeySize() const override { return sizeof(Key); }
  uint32_t ValueSize() const override { return sizeof(Elem) * ctx_.line_size; }
  uint64_t Capacity() const override { return ctx_.n_set * kWarpSize; }
  uint32_t MaxQueryLength() const override { return max_query_length_; }

  void ReserveQueryLength(uint32_t query_length) override {
    CudaCurrentDeviceGuard guard(device_index_);
    if (query_length < max_query_length_) { return; }
    if (max_query_length_ != 0) {
      OF_CUDA_CHECK(hipFree(query_indices_buffer_));
      OF_CUDA_CHECK(hipFree(query_keys_buffer_));
    }
    OF_CUDA_CHECK(hipMalloc(&query_indices_buffer_, query_length * sizeof(uint32_t)));
    OF_CUDA_CHECK(hipMalloc(&query_keys_buffer_, query_length * sizeof(Key)));
    max_query_length_ = query_length;
  }

  CacheOptions::Policy Policy() const override { return CacheOptions::Policy::kLRU; }

  void Test(ep::Stream* stream, uint32_t n_keys, const void* keys, uint32_t* n_missing,
            void* missing_keys, uint32_t* missing_indices) override {
    CHECK_LE(n_keys, max_query_length_);
    auto hip_stream = stream->As<ep::CudaStream>();
    OF_CUDA_CHECK(hipMemsetAsync(n_missing, 0, sizeof(uint32_t), hip_stream->hip_stream()));
    if (n_keys == 0) { return; }
    hip_stream->LaunchKernel(GetKernel<Key, Elem, true>, GetLaunchConfig(n_keys), ctx_, n_keys,
                              static_cast<const Key*>(keys), nullptr, n_missing,
                              static_cast<Key*>(missing_keys), missing_indices);
  }

  void Get(ep::Stream* stream, uint32_t n_keys, const void* keys, void* values, uint32_t* n_missing,
           void* missing_keys, uint32_t* missing_indices) override {
    CHECK_LE(n_keys, max_query_length_);
    auto hip_stream = stream->As<ep::CudaStream>();
    OF_CUDA_CHECK(hipMemsetAsync(n_missing, 0, sizeof(uint32_t), hip_stream->hip_stream()));
    if (n_keys == 0) { return; }
    hip_stream->LaunchKernel(GetKernel<Key, Elem, false>, GetLaunchConfig(n_keys), ctx_, n_keys,
                              static_cast<const Key*>(keys), static_cast<Elem*>(values), n_missing,
                              static_cast<Key*>(missing_keys), missing_indices);
  }

  void Put(ep::Stream* stream, uint32_t n_keys, const void* keys, const void* values,
           uint32_t* n_evicted, void* evicted_keys, void* evicted_values) override {
    CHECK_LE(n_keys, max_query_length_);
    auto hip_stream = stream->As<ep::CudaStream>();
    OF_CUDA_CHECK(hipMemsetAsync(n_evicted, 0, sizeof(uint32_t), hip_stream->hip_stream()));
    if (n_keys == 0) { return; }
    hip_stream->LaunchKernel(PutWithoutEvictingKernel<Key, Elem>, GetLaunchConfig(n_keys), ctx_,
                              n_keys, static_cast<const Key*>(keys),
                              static_cast<const Elem*>(values), n_evicted, query_keys_buffer_,
                              query_indices_buffer_);
    hip_stream->LaunchKernel(EvictKernel<Key, Elem>, GetLaunchConfig(n_keys), ctx_,
                              query_keys_buffer_, query_indices_buffer_,
                              static_cast<const Elem*>(values), n_evicted,
                              static_cast<Key*>(evicted_keys), static_cast<Elem*>(evicted_values));
  }

  void Dump(ep::Stream* stream, uint64_t start_key_index, uint64_t end_key_index,
            uint32_t* n_dumped, void* keys, void* values) override {
    auto hip_stream = stream->As<ep::CudaStream>();
    OF_CUDA_CHECK(hipMemsetAsync(n_dumped, 0, sizeof(uint32_t), hip_stream->hip_stream()));
    const uint64_t max_dump_keys = end_key_index - start_key_index;
    hip_stream->LaunchKernel(
        DumpKernel<Key, Elem>,
        ep::CudaLaunchConfig((max_dump_keys + kNumWarpPerBlock - 1) / kNumWarpPerBlock, kBlockSize,
                             0),
        ctx_, start_key_index, end_key_index, n_dumped, static_cast<Key*>(keys),
        static_cast<Elem*>(values));
  }

  void Clear() override { ClearLruCacheContext<Key, Elem>(&ctx_); }

 private:
  int device_index_;
  uint32_t max_query_length_;
  LruCacheContext<Key, Elem> ctx_;
  uint32_t* query_indices_buffer_;
  Key* query_keys_buffer_;
};

template<typename Key>
std::unique_ptr<Cache> DispatchValueType(const CacheOptions& options) {
  if (options.value_size % sizeof(ulonglong2) == 0) {
    return std::unique_ptr<Cache>(new LruCache<Key, ulonglong2>(options));
  } else if (options.value_size % sizeof(uint64_t) == 0) {
    return std::unique_ptr<Cache>(new LruCache<Key, uint64_t>(options));
  } else if (options.value_size % sizeof(uint32_t) == 0) {
    return std::unique_ptr<Cache>(new LruCache<Key, uint32_t>(options));
  } else if (options.value_size % sizeof(uint16_t) == 0) {
    return std::unique_ptr<Cache>(new LruCache<Key, uint16_t>(options));
  } else {
    return std::unique_ptr<Cache>(new LruCache<Key, uint8_t>(options));
  }
}

std::unique_ptr<Cache> DispatchKeyType(const CacheOptions& options) {
  if (options.key_size == sizeof(uint32_t)) {
    return DispatchValueType<uint32_t>(options);
  } else if (options.key_size == sizeof(uint64_t)) {
    return DispatchValueType<uint64_t>(options);
  } else {
    UNIMPLEMENTED();
    return nullptr;
  }
}

}  // namespace

std::unique_ptr<Cache> NewLruCache(const CacheOptions& options) {
  return std::unique_ptr<Cache>(new LruCache<int64_t, float>(options));
}

}  // namespace embedding

}  // namespace oneflow
