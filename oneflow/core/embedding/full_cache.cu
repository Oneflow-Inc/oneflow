#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/embedding/full_cache.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/embedding/hash_functions.cuh"
#include "oneflow/core/cuda/atomic.cuh"

namespace oneflow {

namespace embedding {

using Key32 = unsigned int;
using Key64 = unsigned long long int;
using Key128 = ulonglong2;

namespace {

template<typename Key, typename Index>
struct alignas(2 * std::max(sizeof(Key), sizeof(Index))) TableEntry {
  Key key;
  Index index;
};

template<typename Key, typename Index>
__device__ bool TryGetOrInsert(Key* entry_key, volatile Index* entry_index, uint64_t* table_size,
                               Key key, uint64_t* out) {
  Key key_hi = (key | 0x1);
  Key key_lo = (key & 0x1);
  Index index_plus_one = 0;
  Key old_entry_key = cuda::atomic::CAS(entry_key, static_cast<Key>(0), key_hi);
  while (index_plus_one == 0) {
    if (old_entry_key == static_cast<Key>(0)) {
      Index index = cuda::atomic::Add(table_size, static_cast<uint64_t>(1));
      index_plus_one = index + 1;
      *entry_index = ((index_plus_one << 1U) | key_lo);
      *out = index_plus_one;
      return true;
    } else if (old_entry_key == key_hi) {
      const Index entry_index_val = *entry_index;
      if (entry_index_val == 0) {
        // do nothing
      } else if ((entry_index_val & 0x1) == key_lo) {
        *out = (entry_index_val >> 1U);
        return true;
      } else {
        return false;
      }
    } else {
      return false;
    }
  }
  return false;
}

template<typename Key, typename Index>
__device__ bool GetOrInsertOne(const size_t capacity, TableEntry<Key, Index>* table,
                               uint64_t* table_size, Key key, size_t hash, uint64_t* out) {
  const size_t start_idx = hash % capacity;
  for (size_t count = 0; count < capacity; ++count) {
    const size_t idx = (start_idx + count) % capacity;
    Key* entry_key = &table[idx].key;
    Index* entry_index = &table[idx].index;
    if (TryGetOrInsert<Key, Index>(entry_key, entry_index, table_size, key, out)) { return true; }
  }
  return false;
}

template<typename Key, typename Index>
__device__ bool GetOne(const size_t capacity, TableEntry<Key, Index>* table, Key key, size_t hash,
                       uint64_t* out) {
  const size_t start_idx = hash % capacity;
  for (size_t count = 0; count < capacity; ++count) {
    const size_t idx = (start_idx + count) % capacity;
    TableEntry<Key, Index> entry = table[idx];
    Key key_hi = (key | 0x1);
    Key key_lo = (key & 0x1);
    if (entry.key == 0) { break; }
    if (entry.key == key_hi) {
      if ((entry.index & 0x1) == key_lo) {
        *out = (entry.index >> 1U);
        return true;
      }
    }
  }
  *out = 0;
  return false;
}

template<typename Key, typename Index>
__global__ void OrdinalEncodeKernel(uint64_t capacity, TableEntry<Key, Index>* table,
                                    uint64_t* table_size, uint32_t num_keys, const Key* keys,
                                    uint64_t* context) {
  CUDA_1D_KERNEL_LOOP(i, num_keys) {
    Key key = keys[i];
    uint64_t hash = FullCacheHash()(key);
    bool success = GetOrInsertOne<Key, Index>(capacity, table, table_size, key, hash, context + i);
    assert(success);
  }
}

template<typename Key, typename Index>
__global__ void OrdinalEncodeLookupKernel(uint64_t capacity, TableEntry<Key, Index>* table,
                                          uint32_t num_keys, const Key* keys, uint64_t* context) {
  CUDA_1D_KERNEL_LOOP(i, num_keys) {
    Key key = keys[i];
    uint64_t hash = FullCacheHash()(key);
    GetOne<Key, Index>(capacity, table, key, hash, context + i);
  }
}

template<typename Key, typename Index>
__global__ void OrdinalEncodeDumpKernel(const TableEntry<Key, Index>* table,
                                        uint64_t start_key_index, uint64_t end_key_index,
                                        uint32_t* n_dumped, Key* keys, uint64_t* context) {
  CUDA_1D_KERNEL_LOOP(i, (end_key_index - start_key_index)) {
    TableEntry<Key, Index> entry = table[i + start_key_index];
    if (entry.index != 0) {
      uint32_t index = cuda::atomic::Add(n_dumped, static_cast<uint32_t>(1));
      keys[index] = ((entry.key ^ 0x1) | (entry.index & 0x1));
      context[index] = (entry.index >> 1U);
    }
  }
}

template<typename Key, typename Elem, bool return_value>
__global__ void LookupKernel(uint32_t value_length, const Elem* cache_values,
                             uint32_t values_elem_cnt, const Key* keys, const uint64_t* context,
                             Elem* values, uint32_t* n_missing, Key* missing_keys,
                             uint32_t* missing_indices) {
  CUDA_1D_KERNEL_LOOP(i, values_elem_cnt) {
    const uint64_t key_id = i / value_length;
    const uint64_t ctx = context[key_id];
    const uint64_t row_id = ctx - 1;
    const uint64_t col_id = i - key_id * value_length;
    if (ctx == 0) {
      const Key missing_key = keys[key_id];
      if (col_id == 0) {
        const uint32_t old_n_missing = cuda::atomic::Add(n_missing, static_cast<uint32_t>(1));
        missing_keys[old_n_missing] = missing_key;
        missing_indices[old_n_missing] = key_id;
      }
      continue;
    }
    if (return_value) { values[i] = cache_values[row_id * value_length + col_id]; }
  }
}

template<typename Elem>
__global__ void UpdateKernel(uint32_t value_length, Elem* cache_values, uint32_t values_elem_cnt,
                             const uint64_t* context, const Elem* values) {
  CUDA_1D_KERNEL_LOOP(i, values_elem_cnt) {
    const uint64_t key_id = i / value_length;
    const uint64_t ctx = context[key_id];
    if (ctx == 0) { continue; }
    const uint64_t row_id = ctx - 1;
    const uint64_t col_id = i - key_id * value_length;
    const Elem elem = values[i];
    cache_values[row_id * value_length + col_id] = elem;
  }
}

template<typename Key, typename Elem>
__global__ void DumpValueKernel(uint32_t value_length, const uint32_t* n_dumped,
                                const uint64_t* context, const Elem* cache_values, Elem* values) {
  CUDA_1D_KERNEL_LOOP(i, *n_dumped * value_length) {
    const uint64_t key_id = i / value_length;
    const uint64_t ctx = context[key_id];
    const uint64_t row_id = ctx - 1;
    const uint64_t col_id = i - key_id * value_length;
    values[i] = cache_values[row_id * value_length + col_id];
  }
}

template<typename Key, typename Index>
class OrdinalEncoder {
 public:
  OF_DISALLOW_COPY_AND_MOVE(OrdinalEncoder);
  explicit OrdinalEncoder(uint64_t capacity, float load_factor)
      : capacity_(capacity), table_capacity_(capacity / load_factor) {
    OF_CUDA_CHECK(hipGetDevice(&device_index_));
    OF_CUDA_CHECK(hipMalloc(&table_size_, sizeof(uint64_t)));
    OF_CUDA_CHECK(hipHostMalloc(&table_size_host_, sizeof(uint64_t)));
    OF_CUDA_CHECK(hipMalloc(&table_, table_capacity_ * sizeof(TableEntry<Key, Index>)));
    Clear();
  }
  ~OrdinalEncoder() {
    CudaCurrentDeviceGuard guard(device_index_);
    OF_CUDA_CHECK(hipFree(table_size_));
    OF_CUDA_CHECK(hipHostFree(table_size_host_));
    OF_CUDA_CHECK(hipFree(table_));
  }

  template<bool insert>
  void Encode(ep::Stream* stream, uint32_t num_keys, const Key* keys, uint64_t* context) {
    if (insert) {
      RUN_CUDA_KERNEL((OrdinalEncodeKernel<Key, uint64_t>), stream, num_keys, table_capacity_,
                      table_, table_size_, num_keys, keys, context);
      OF_CUDA_CHECK(hipMemcpyAsync(table_size_host_, table_size_, sizeof(uint64_t),
                                    hipMemcpyDefault,
                                    stream->As<ep::CudaStream>()->hip_stream()));
      CHECK_JUST(stream->Sync());
      CHECK_LT(*table_size_host_, capacity_)
          << "The number of key is larger than cache size, please enlarge cache_memory_budget. ";
    } else {
      RUN_CUDA_KERNEL((OrdinalEncodeLookupKernel<Key, uint64_t>), stream, num_keys, table_capacity_,
                      table_, num_keys, keys, context);
    }
  }

  void Dump(ep::Stream* stream, uint64_t start_key_index, uint64_t end_key_index,
            uint32_t* n_dumped, Key* keys, uint64_t* context) {
    OF_CUDA_CHECK(hipMemsetAsync(n_dumped, 0, sizeof(uint32_t),
                                  stream->As<ep::CudaStream>()->hip_stream()));
    RUN_CUDA_KERNEL((OrdinalEncodeDumpKernel<Key, uint64_t>), stream,
                    end_key_index - start_key_index, table_, start_key_index, end_key_index,
                    n_dumped, keys, context);
  }

  void Clear() {
    OF_CUDA_CHECK(hipMemset(table_size_, 0, sizeof(uint64_t)));
    OF_CUDA_CHECK(hipMemset(table_, 0, table_capacity_ * sizeof(TableEntry<Key, Index>)));
  }

  uint64_t TableCapacity() const { return table_capacity_; }

 private:
  int device_index_{};
  TableEntry<Key, Index>* table_;
  uint64_t capacity_;
  uint64_t table_capacity_;
  uint64_t* table_size_{};
  uint64_t* table_size_host_{};
};

template<typename Key, typename Elem>
class CacheImpl : public Cache {
 public:
  OF_DISALLOW_COPY_AND_MOVE(CacheImpl);
  explicit CacheImpl(const CacheOptions& options)
      : encoder_(options.capacity, options.load_factor),
        device_index_(-1),
        options_(options),
        max_query_length_(0) {
    OF_CUDA_CHECK(hipGetDevice(&device_index_));
    const uint64_t values_size = options.capacity * options.value_size;
    if (options.value_memory_kind == CacheOptions::MemoryKind::kDevice) {
      OF_CUDA_CHECK(hipMalloc(&values_, values_size));
    } else if (options.value_memory_kind == CacheOptions::MemoryKind::kHost) {
      OF_CUDA_CHECK(
          NumaAwareCudaMallocHost(device_index_, reinterpret_cast<void**>(&values_), values_size));
    } else {
      UNIMPLEMENTED();
    }
    num_elem_per_value_ = options_.value_size / sizeof(Elem);
  }
  ~CacheImpl() {
    CudaCurrentDeviceGuard guard(device_index_);
    if (options_.value_memory_kind == CacheOptions::MemoryKind::kDevice) {
      OF_CUDA_CHECK(hipFree(values_));
    } else if (options_.value_memory_kind == CacheOptions::MemoryKind::kHost) {
      OF_CUDA_CHECK(hipHostFree(values_));
    } else {
      UNIMPLEMENTED();
    }
    if (max_query_length_ > 0) { OF_CUDA_CHECK(hipFree(encoding_buffer_)); }
  }

  uint64_t Capacity() const override { return options_.capacity; }
  uint64_t DumpCapacity() const override { return encoder_.TableCapacity(); }
  uint32_t KeySize() const override { return options_.key_size; }

  uint32_t ValueSize() const override { return options_.value_size; }

  uint32_t MaxQueryLength() const override { return max_query_length_; }

  void ReserveQueryLength(uint32_t query_length) override {
    CudaCurrentDeviceGuard guard(device_index_);
    if (query_length <= max_query_length_) { return; }
    if (max_query_length_ > 0) { OF_CUDA_CHECK(hipFree(encoding_buffer_)); }
    OF_CUDA_CHECK(hipMalloc(&encoding_buffer_, query_length * sizeof(uint64_t)));
    max_query_length_ = query_length;
  }

  CacheOptions::Policy Policy() const override { return CacheOptions::Policy::kFull; }

  void Test(ep::Stream* stream, uint32_t n_keys, const void* keys, uint32_t* n_missing,
            void* missing_keys, uint32_t* missing_indices) override;

  void Get(ep::Stream* stream, uint32_t n_keys, const void* keys, void* values, uint32_t* n_missing,
           void* missing_keys, uint32_t* missing_indices) override;

  void Put(ep::Stream* stream, uint32_t n_keys, const void* keys, const void* values,
           uint32_t* n_evicted, void* evicted_keys, void* evicted_values) override;

  void Dump(ep::Stream* stream, uint64_t start_key_index, uint64_t end_key_index,
            uint32_t* n_dumped, void* keys, void* values) override;

  void Clear() override;

 private:
  OrdinalEncoder<Key, uint64_t> encoder_;
  int device_index_;
  uint32_t num_elem_per_value_{};
  Elem* values_;
  uint64_t* encoding_buffer_{};
  CacheOptions options_;
  uint32_t max_query_length_;
};

template<typename Key, typename Elem>
void CacheImpl<Key, Elem>::Test(ep::Stream* stream, uint32_t n_keys, const void* keys,
                                uint32_t* n_missing, void* missing_keys,
                                uint32_t* missing_indices) {
  OF_CUDA_CHECK(
      hipMemsetAsync(n_missing, 0, sizeof(uint32_t), stream->As<ep::CudaStream>()->hip_stream()));
  if (n_keys == 0) { return; }
  CHECK_LE(n_keys, max_query_length_);
  encoder_.template Encode<false>(stream, n_keys, static_cast<const Key*>(keys), encoding_buffer_);
  const uint32_t values_elem_cnt = n_keys * num_elem_per_value_;
  RUN_CUDA_KERNEL((LookupKernel<Key, Elem, false>), stream, values_elem_cnt, num_elem_per_value_,
                  values_, values_elem_cnt, static_cast<const Key*>(keys), encoding_buffer_,
                  nullptr, n_missing, static_cast<Key*>(missing_keys), missing_indices);
}

template<typename Key, typename Elem>
void CacheImpl<Key, Elem>::Get(ep::Stream* stream, uint32_t n_keys, const void* keys, void* values,
                               uint32_t* n_missing, void* missing_keys, uint32_t* missing_indices) {
  OF_CUDA_CHECK(
      hipMemsetAsync(n_missing, 0, sizeof(uint32_t), stream->As<ep::CudaStream>()->hip_stream()));
  if (n_keys == 0) { return; }
  CHECK_LE(n_keys, max_query_length_);
  encoder_.template Encode<false>(stream, n_keys, static_cast<const Key*>(keys), encoding_buffer_);
  const uint32_t values_elem_cnt = n_keys * num_elem_per_value_;
  RUN_CUDA_KERNEL((LookupKernel<Key, Elem, true>), stream, values_elem_cnt, num_elem_per_value_,
                  values_, values_elem_cnt, static_cast<const Key*>(keys), encoding_buffer_,
                  static_cast<Elem*>(values), n_missing, static_cast<Key*>(missing_keys),
                  missing_indices);
}

template<typename Key, typename Elem>
void CacheImpl<Key, Elem>::Put(ep::Stream* stream, uint32_t n_keys, const void* keys,
                               const void* values, uint32_t* n_evicted, void* evicted_keys,
                               void* evicted_values) {
  OF_CUDA_CHECK(
      hipMemsetAsync(n_evicted, 0, sizeof(uint32_t), stream->As<ep::CudaStream>()->hip_stream()));
  if (n_keys == 0) { return; }
  CHECK_LE(n_keys, max_query_length_);
  encoder_.template Encode<true>(stream, n_keys, static_cast<const Key*>(keys), encoding_buffer_);
  const uint32_t values_elem_cnt = n_keys * num_elem_per_value_;
  RUN_CUDA_KERNEL((UpdateKernel<Elem>), stream, values_elem_cnt, num_elem_per_value_, values_,
                  values_elem_cnt, encoding_buffer_, static_cast<const Elem*>(values));
}

template<typename Key, typename Elem>
void CacheImpl<Key, Elem>::Dump(ep::Stream* stream, uint64_t start_key_index,
                                uint64_t end_key_index, uint32_t* n_dumped, void* keys,
                                void* values) {
  encoder_.Dump(stream, start_key_index, end_key_index, n_dumped, static_cast<Key*>(keys),
                encoding_buffer_);
  RUN_CUDA_KERNEL((DumpValueKernel<Key, Elem>), stream,
                  num_elem_per_value_ * (end_key_index - start_key_index), num_elem_per_value_,
                  n_dumped, encoding_buffer_, values_, static_cast<Elem*>(values));
}

template<typename Key, typename Elem>
void CacheImpl<Key, Elem>::Clear() {
  encoder_.Clear();
}

template<typename Key>
std::unique_ptr<Cache> DispatchValueType(const CacheOptions& options) {
  if (options.value_size % sizeof(ulonglong2) == 0) {
    return std::unique_ptr<Cache>(new CacheImpl<Key, ulonglong2>(options));
  } else if (options.value_size % sizeof(uint64_t) == 0) {
    return std::unique_ptr<Cache>(new CacheImpl<Key, uint64_t>(options));
  } else if (options.value_size % sizeof(uint32_t) == 0) {
    return std::unique_ptr<Cache>(new CacheImpl<Key, uint32_t>(options));
  } else if (options.value_size % sizeof(uint16_t) == 0) {
    return std::unique_ptr<Cache>(new CacheImpl<Key, uint16_t>(options));
  } else {
    return std::unique_ptr<Cache>(new CacheImpl<Key, uint8_t>(options));
  }
}

std::unique_ptr<Cache> DispatchKeyType(const CacheOptions& options) {
  if (options.key_size == sizeof(Key32)) {
    return DispatchValueType<Key32>(options);
  } else if (options.key_size == sizeof(Key64)) {
    return DispatchValueType<Key64>(options);
  } else {
    UNIMPLEMENTED();
    return nullptr;
  }
}

}  // namespace

std::unique_ptr<Cache> NewFullCache(const CacheOptions& options) {
  return DispatchKeyType(options);
}

}  // namespace embedding

}  // namespace oneflow
