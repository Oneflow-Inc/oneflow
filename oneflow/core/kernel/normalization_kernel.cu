#include "oneflow/core/kernel/normalization_kernel.h"

namespace oneflow {

template<>
void NormalizationKernel<DeviceType::kGPU, float>::NormalizationCudnnForward(
    const KernelCtx& ctx,
    const std::function<Blob*(const std::string&)>& BnInOp2Blob) const {
  const float* in = BnInOp2Blob("in")->dptr<float>();
  const float* gamma = BnInOp2Blob("gamma")->dptr<float>();
  const float* beta = BnInOp2Blob("beta")->dptr<float>();
  float* out = BnInOp2Blob("out")->mut_dptr<float>();
  float* moving_mean = BnInOp2Blob("moving_mean")->mut_dptr<float>();
  float* moving_variance = BnInOp2Blob("moving_variance")->mut_dptr<float>();
  double epsilon = this->op_conf().normalization_conf().epsilon();
  if (Global<JobDesc>::Get()->IsTrain()) {
    InitMovingMeanAndMovingVariance(ctx, BnInOp2Blob, false);
    double momentum = this->op_conf().normalization_conf().momentum();
    CudaCheck(hipdnnBatchNormalizationForwardTraining(
        ctx.device_ctx->cudnn_handle(),
        normalization_ctx_->cudnn_batch_norm_mode(), OnePtr<float>::value,
        ZeroPtr<float>::value, normalization_ctx_->cudnn_in_tensor_desc(), in,
        normalization_ctx_->cudnn_in_tensor_desc(), out,
        normalization_ctx_->cudnn_param_tensor_desc(), gamma, beta,
        1 - momentum, moving_mean, moving_variance, epsilon,
        BnInOp2Blob("cache_mean_for_cudnn_bw")->mut_dptr<float>(),
        BnInOp2Blob("cache_inv_variance_for_cudnn_bw")->mut_dptr<float>()));
  } else {
    CudaCheck(hipdnnBatchNormalizationForwardInference(
        ctx.device_ctx->cudnn_handle(), HIPDNN_BATCHNORM_SPATIAL,
        OnePtr<float>::value, ZeroPtr<float>::value,
        normalization_ctx_->cudnn_in_tensor_desc(), in,
        normalization_ctx_->cudnn_in_tensor_desc(), out,
        normalization_ctx_->cudnn_param_tensor_desc(), gamma, beta, moving_mean,
        moving_variance, epsilon));
  }
}
template<>
void NormalizationKernel<DeviceType::kGPU, float>::NormalizationCudnnBackward(
    const KernelCtx& ctx,
    const std::function<Blob*(const std::string&)>& BnInOp2Blob) const {
  const hipdnnTensorDescriptor_t& in_desc =
      normalization_ctx_->cudnn_in_tensor_desc();
  CudaCheck(hipdnnBatchNormalizationBackward(
      ctx.device_ctx->cudnn_handle(),
      normalization_ctx_->cudnn_batch_norm_mode(), OnePtr<float>::value,
      ZeroPtr<float>::value, OnePtr<float>::value, ZeroPtr<float>::value,
      in_desc, BnInOp2Blob("in")->dptr<float>(), in_desc,
      BnInOp2Blob(GenDiffBn("out"))->dptr<float>(), in_desc,
      BnInOp2Blob(GenDiffBn("in"))->mut_dptr<float>(),
      normalization_ctx_->cudnn_param_tensor_desc(),
      BnInOp2Blob("gamma")->dptr<float>(),
      BnInOp2Blob(GenDiffBn("gamma"))->mut_dptr<float>(),
      BnInOp2Blob(GenDiffBn("beta"))->mut_dptr<float>(),
      static_cast<double>(this->op_conf().normalization_conf().epsilon()),
      BnInOp2Blob("cache_mean_for_cudnn_bw")->dptr<float>(),
      BnInOp2Blob("cache_inv_variance_for_cudnn_bw")->dptr<float>()));
}

}  // namespace oneflow
