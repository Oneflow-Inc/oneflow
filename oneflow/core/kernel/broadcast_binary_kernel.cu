#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/broadcast_binary_kernel.h"
#include "oneflow/core/ndarray/xpu_broadcast_ndarray.h"
#include "oneflow/core/ndarray/xpu_binary_func_ndarray.h"
#include "oneflow/core/ndarray/gpu_ndarray_assign.h"
#include "oneflow/core/ndarray/binary_func.h"

namespace oneflow {

namespace {

template<typename T, int NDIMS, const T (*binary_func)(const T, const T)>
__global__ void GpuBroadcastBinaryFunc(XpuVarNdarray<T> y, const XpuVarNdarray<const T> a,
                                       const XpuVarNdarray<const T> b) {
  XpuBroadcastNdarray<const T> a_broadcasted(y.shape(), a);
  XpuBroadcastNdarray<const T> b_broadcasted(y.shape(), b);
  XpuBinaryFuncNdarray<const T, binary_func> binary_func_ndarray(a_broadcasted, b_broadcasted);
  GpuNdArrayAssign<NDIMS>(&y, binary_func_ndarray);
}

}  // namespace

template<typename T, int NDIMS, const T (*binary_func)(const T, const T)>
struct BroadcastBinaryFunc<DeviceType::kGPU, T, NDIMS, binary_func> final {
  static void Invoke(DeviceCtx* ctx, XpuVarNdarray<T>&& y, const XpuVarNdarray<const T>& a,
                     const XpuVarNdarray<const T>& b) {
    size_t n = y.shape().ElemNum();
    GpuBroadcastBinaryFunc<T, NDIMS, binary_func>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(y, a, b);
  }
};

#define INSTANTIATE_BROADCAST_BINARY_FUNC(dtype_pair, NDIMS, binary_func)                    \
  template struct BroadcastBinaryFunc<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype_pair), NDIMS, \
                                      binary_func>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_BROADCAST_BINARY_FUNC, ARITHMETIC_DATA_TYPE_SEQ,
                                 DIM_SEQ, ARITHMETIC_BINARY_FUNC_SEQ)
}  // namespace oneflow
