#include "hip/hip_runtime.h"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/common/util.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/sparse_cross_entropy_loss_kernel.h"

namespace oneflow {

namespace {

template<typename PredType, typename LabelType>
__global__ void SparseCrossEntropyLossForwardGpu(const int64_t instance_num,
                                                 const int64_t num_of_classes,
                                                 const PredType* prediction,
                                                 const LabelType* labels,
                                                 PredType* loss) {
  CUDA_1D_KERNEL_LOOP(i, instance_num) {
    int64_t label = static_cast<int64_t>(labels[i]);
    assert(label >= 0);
    assert(label < num_of_classes);
    loss[i] = -SAFE_LOG(prediction[i * num_of_classes + label]);
  }
}

template<typename PredType, typename LabelType>
__global__ void SparseCrossEntropyLossBackwardGpu(const int64_t instance_num,
                                                  const int64_t num_of_classes,
                                                  const PredType* prediction,
                                                  const LabelType* labels,
                                                  PredType* prediction_diff) {
  CUDA_1D_KERNEL_LOOP(i, instance_num) {
    int64_t label = static_cast<int64_t>(labels[i]);
    PredType prob = prediction[i * num_of_classes + label];
    prediction_diff[i * num_of_classes + label] =
        -1 / MAX_WITH_LOG_THRESHOLD(prob);
  }
}

}  // namespace

template<typename PredType, typename LabelType>
struct SparseCrossEntropyLossKernelUtil<DeviceType::kGPU, PredType, LabelType> {
  static void Forward(DeviceCtx* ctx, const int64_t instance_num,
                      const int64_t num_of_classes, const PredType* prediction,
                      const LabelType* labels, PredType* loss) {
    SparseCrossEntropyLossForwardGpu<PredType>
        <<<BlocksNum4ThreadsNum(instance_num), kCudaThreadsNumPerBlock, 0,
           ctx->hip_stream()>>>(instance_num, num_of_classes, prediction,
                                 labels, loss);
  }

  static void Backward(DeviceCtx* ctx, const int64_t instance_num,
                       const int64_t num_of_classes, const PredType* prediction,
                       const LabelType* labels, PredType* prediction_diff) {
    SparseCrossEntropyLossBackwardGpu<PredType>
        <<<BlocksNum4ThreadsNum(instance_num), kCudaThreadsNumPerBlock, 0,
           ctx->hip_stream()>>>(instance_num, num_of_classes, prediction,
                                 labels, prediction_diff);
  }
};

#define MAKE_ENTRY(data_type_pair, label_type_pair)       \
  template struct SparseCrossEntropyLossKernelUtil<       \
      DeviceType::kGPU, OF_PP_PAIR_FIRST(data_type_pair), \
      OF_PP_PAIR_FIRST(label_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_ENTRY, FLOATING_DATA_TYPE_SEQ,
                                 INT_DATA_TYPE_SEQ)

}  // namespace oneflow
