#include "oneflow/core/kernel/layer_norm_kernel.h"

namespace oneflow {

namespace {

class LayerNormCudnnBnCtx final {
 public:
  LayerNormCudnnBnCtx(const Shape& data_shape, const Shape& param_shape, DataType data_type) {
    const int64_t cudnn_c = param_shape.elem_cnt();
    CHECK_EQ(data_shape.elem_cnt() % cudnn_c, 0);
    const int64_t cudnn_w = data_shape.elem_cnt() / cudnn_c;
    CHECK_LT(cudnn_c, MaxVal<int32_t>::value);
    CHECK_LT(cudnn_w, MaxVal<int32_t>::value);
    data_tensor_desc_.reset(new CudnnTensorDesc(HIPDNN_TENSOR_NCHW, data_type, 1,
                                                static_cast<int32_t>(cudnn_c), 1,
                                                static_cast<int32_t>(cudnn_w)));
    param_tensor_desc_.reset(
        new CudnnTensorDesc(HIPDNN_TENSOR_NCHW, data_type, 1, static_cast<int32_t>(cudnn_c), 1, 1));
#if (CUDNN_VERSION >= 7000)
    mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;
#else
    mode_ = HIPDNN_BATCHNORM_SPATIAL;
#endif
  }
  ~LayerNormCudnnBnCtx() = default;

  const hipdnnTensorDescriptor_t& data_tensor_desc() const { return data_tensor_desc_->Get(); }
  const hipdnnTensorDescriptor_t& param_tensor_desc() const { return param_tensor_desc_->Get(); }
  hipdnnBatchNormMode_t mode() const { return mode_; };

 private:
  std::unique_ptr<CudnnTensorDesc> data_tensor_desc_;
  std::unique_ptr<CudnnTensorDesc> param_tensor_desc_;
  hipdnnBatchNormMode_t mode_;
};

}  // namespace

template<typename T>
struct LayerNormKernelUtil<DeviceType::kGPU, T> {
  static void NormalizeForward(const DeviceCtx* ctx, const Blob* in, const Blob* scale,
                               const Blob* bias, double epsilon, Blob* out, Blob* mean,
                               Blob* inv_variance);
  static void NormalizeBackward(const DeviceCtx* ctx, const Blob* in, const Blob* scale,
                                const Blob* mean, const Blob* inv_variance, const Blob* out_diff,
                                double epsilon, Blob* in_diff, Blob* scale_diff, Blob* bias_diff);
};

template<typename T>
void LayerNormKernelUtil<DeviceType::kGPU, T>::NormalizeForward(const DeviceCtx* ctx,
                                                                const Blob* in, const Blob* scale,
                                                                const Blob* bias, double epsilon,
                                                                Blob* out, Blob* mean,
                                                                Blob* inv_variance) {
  CHECK_GE(epsilon, HIPDNN_BN_MIN_EPSILON);
  LayerNormCudnnBnCtx bn_ctx(in->static_shape(), mean->shape(), in->data_type());
  CudaCheck(hipdnnBatchNormalizationForwardTraining(
      ctx->cudnn_handle(), bn_ctx.mode(), OnePtr<T>::value, ZeroPtr<T>::value,
      bn_ctx.data_tensor_desc(), in->dptr<T>(), bn_ctx.data_tensor_desc(), out->mut_dptr<T>(),
      bn_ctx.param_tensor_desc(), scale->dptr<T>(), bias->dptr<T>(), 1.0, nullptr, nullptr, epsilon,
      mean->mut_dptr<T>(), inv_variance->mut_dptr<T>()));
}

template<typename T>
void LayerNormKernelUtil<DeviceType::kGPU, T>::NormalizeBackward(
    const DeviceCtx* ctx, const Blob* in, const Blob* scale, const Blob* mean,
    const Blob* inv_variance, const Blob* out_diff, double epsilon, Blob* in_diff, Blob* scale_diff,
    Blob* bias_diff) {
  CHECK_GE(epsilon, HIPDNN_BN_MIN_EPSILON);
  LayerNormCudnnBnCtx bn_ctx(in->static_shape(), mean->shape(), in->data_type());
  CudaCheck(hipdnnBatchNormalizationBackward(
      ctx->cudnn_handle(), bn_ctx.mode(), OnePtr<T>::value, ZeroPtr<T>::value, OnePtr<T>::value,
      ZeroPtr<T>::value, bn_ctx.data_tensor_desc(), in->dptr<T>(), bn_ctx.data_tensor_desc(),
      out_diff->dptr<T>(), bn_ctx.data_tensor_desc(), in_diff->mut_dptr<T>(),
      bn_ctx.param_tensor_desc(), scale->dptr<T>(), scale_diff->mut_dptr<T>(),
      bias_diff->mut_dptr<T>(), epsilon, mean->dptr<T>(), inv_variance->dptr<T>()));
}

#define INSTANTIATE_LAYER_NORM_KERNEL_UTIL_GPU(type_cpp, type_proto) \
  template struct LayerNormKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_LAYER_NORM_KERNEL_UTIL_GPU, FLOATING_DATA_TYPE_SEQ)
#undef INSTANTIATE_LAYER_NORM_KERNEL_UTIL_GPU

}  // namespace oneflow
