#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/dropout_kernel.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/common/preprocessor.h"

namespace oneflow {

namespace {

template<typename T>
__global__ typename std::enable_if<!IsFloat16<T>::value>::type MaskAndScaleGpu(
    const int64_t n, float scale, const T* x, const int8_t* mask, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = x[i] * static_cast<T>(mask[i]) * scale; }
}

template<>
__global__ void MaskAndScaleGpu<half>(const int64_t n, float scale, const half* x,
                                      const int8_t* mask, half* y) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  half h_scale = __float2half(scale);
  CUDA_1D_KERNEL_LOOP(i, n) {
    half one_or_zero = mask[i];
    y[i] = __hmul(__hmul(x[i], one_or_zero), h_scale);
  }
#else
  printf("use half need nvcc arch >= 530");
  assert(false);
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)*/
}

}  // namespace

template<typename T>
struct DropoutKernelUtil<DeviceType::kGPU, T, typename std::enable_if<!IsFloat16<T>::value>::type>
    final {
  static void MaskAndScale(DeviceCtx* ctx, const int64_t n, float scale, const T* x,
                           const int8_t* mask, T* y) {
    MaskAndScaleGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, scale, x, mask, y);
  }
};

template<typename T>
struct DropoutKernelUtil<DeviceType::kGPU, T, typename std::enable_if<IsFloat16<T>::value>::type>
    final {
  static void MaskAndScale(DeviceCtx* ctx, const int64_t n, float scale, const T* x,
                           const int8_t* mask, T* y) {
    MaskAndScaleGpu<half>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            n, scale, reinterpret_cast<const half*>(x), mask, reinterpret_cast<half*>(y));
  }
};

#define INITIATE_DROPOUT_KERNEL_UTIL_GPU(T, type_proto) \
  template struct DropoutKernelUtil<DeviceType::kGPU, T>;
OF_PP_FOR_EACH_TUPLE(INITIATE_DROPOUT_KERNEL_UTIL_GPU,
                     ARITHMETIC_DATA_TYPE_SEQ FLOAT16_DATA_TYPE_SEQ);
#undef INITIATE_DROPOUT_KERNEL_UTIL_GPU

}  // namespace oneflow
