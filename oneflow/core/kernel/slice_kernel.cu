#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/slice_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void SliceForwardGpu(const int64_t n, const int64_t* offset, const T* entire, T* slice) {
  CUDA_1D_KERNEL_LOOP(i, n) { slice[i] = entire[offset[i]]; }
}

template<typename T>
__global__ void SliceBackwardGpu(const int64_t n, const int64_t* offset, const T* slice,
                                 T* entire) {
  CUDA_1D_KERNEL_LOOP(i, n) { entire[offset[i]] /= slice[i]; }
}

}  // namespace

template<typename T>
void SliceKernel<DeviceType::kGPU, T>::ForwardDataContent(
    const KernelCtx& ctx, std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  const Blob* in_blob = BnInOp2Blob("in");
  const Blob* offset_blob = BnInOp2Blob("out_to_in_offset");
  Blob* out_blob = BnInOp2Blob("out");
  const int64_t num_output = out_blob->shape().elem_cnt();
  SliceForwardGpu<T><<<BlocksNum4ThreadsNum(num_output), kCudaThreadsNumPerBlock, 0,
                       ctx.device_ctx->hip_stream()>>>(
      num_output, offset_blob->dptr<int64_t>(), in_blob->dptr<T>(), out_blob->mut_dptr<T>());
}

template<typename T>
void SliceKernel<DeviceType::kGPU, T>::BackwardDataContent(
    const KernelCtx& ctx, std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  const Blob* out_diff_blob = BnInOp2Blob(GenDiffBn("out"));
  const Blob* offset_blob = BnInOp2Blob("out_to_in_offset");
  Blob* in_diff_blob = BnInOp2Blob(GenDiffBn("in"));
  const int64_t num_output = out_diff_blob->shape().elem_cnt();
  SliceBackwardGpu<T><<<BlocksNum4ThreadsNum(num_output), kCudaThreadsNumPerBlock, 0,
                        ctx.device_ctx->hip_stream()>>>(num_output, offset_blob->dptr<int64_t>(),
                                                         out_diff_blob->dptr<T>(),
                                                         in_diff_blob->mut_dptr<T>());
}

template<typename T>
void SliceKernel<DeviceType::kGPU, T>::InitConstBufBlobs(
    DeviceCtx* ctx, std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  InitOut2InOffsetFromHost(ctx, BnInOp2Blob("in"), BnInOp2Blob("out_to_in_offset"));
}

template<typename T>
void SliceKernel<DeviceType::kGPU, T>::InitOut2InOffsetFromHost(DeviceCtx* ctx, const Blob* in_blob,
                                                                Blob* blob) const {
  const SliceOpConf& conf = op_conf().slice_conf();
  BEFORE_CPU_INITIALIZE();
  int64_t* host_blob_ptr = host_blob->mut_dptr<int64_t>();
  FOR_RANGE(size_t, i, 0, host_blob->shape().elem_cnt()) {
    size_t offset = 0;
    FOR_RANGE(size_t, j, 0, host_blob->shape().NumAxes()) {
      const size_t dim_idx = i / host_blob->shape().Count(j + 1);
      const size_t dim_len = host_blob->shape().At(j);
      int64_t start = 0;
      int64_t end = dim_len;
      int64_t stride = 1;
      if (j > 0) {
        const DimSliceConf& dim_slice_conf = conf.dim_slice_conf(j - 1);
        if (dim_slice_conf.has_start()) { start = dim_slice_conf.start(); }
        if (dim_slice_conf.has_end()) { end = dim_slice_conf.end(); }
        stride = dim_slice_conf.stride();
      }
      if (start < 0) { start += dim_len; }
      if (end < 0) { end += dim_len; }
      offset += (start + dim_idx * stride) * in_blob->shape().Count(j + 1);
    }
    host_blob_ptr[i] = offset;
  }
  AFTER_CPU_INITIALIZE();
}

#define INSTANTIATE_GPU_SLICE_KERNEL(type_cpp, type_proto) \
  template struct SliceKernel<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GPU_SLICE_KERNEL, ARITHMETIC_DATA_TYPE_SEQ)

}  // namespace oneflow
