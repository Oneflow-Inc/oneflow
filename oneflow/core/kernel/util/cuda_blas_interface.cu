#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/kernel/util/cuda_blas_interface.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/register/blob.h"
#include "oneflow/core/kernel/util/cuda_half_util.h"

namespace oneflow {

namespace {

hipblasOperation_t CblasTrans2CublasTrans(CBLAS_TRANSPOSE trans) {
  hipblasOperation_t cublas_trans;
  if (trans == CBLAS_TRANSPOSE::CblasNoTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_N;
  } else if (trans == CBLAS_TRANSPOSE::CblasTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_T;
  } else if (trans == CBLAS_TRANSPOSE::CblasConjTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_C;
  } else {
    // do nothing
  }
  return cublas_trans;
}

std::tuple<int, int, int, hipblasOperation_t, hipblasOperation_t> PrepareToCallCublasGemm(
    enum CBLAS_TRANSPOSE trans_a, enum CBLAS_TRANSPOSE trans_b, const int m, const int n,
    const int k) {
  int lda = (trans_a == CblasNoTrans) ? k : m;
  int ldb = (trans_b == CblasNoTrans) ? n : k;
  int ldc = n;
  hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
  hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
  return std::make_tuple(lda, ldb, ldc, cublas_trans_a, cublas_trans_b);
}

template<typename T>
void Gemm(DeviceCtx* ctx, const enum CBLAS_ORDER order, enum CBLAS_TRANSPOSE trans_a,
          enum CBLAS_TRANSPOSE trans_b, const int m, const int n, const int k, const double alpha,
          const T* a, const T* b, const double beta, T* c) {
  int lda, ldb, ldc;
  hipblasOperation_t cublas_trans_a, cublas_trans_b;
  std::tie(lda, ldb, ldc, cublas_trans_a, cublas_trans_b) =
      PrepareToCallCublasGemm(trans_a, trans_b, m, n, k);

  hipblasHandle_t handle;
  if (std::is_same<T, half>::value) {
    handle = ctx->cublas_tensor_op_math_handle();
  } else {
    handle = ctx->cublas_pmh_handle();
  }
  const T alpha_val = static_cast<T>(alpha);
  const T beta_val = static_cast<T>(beta);
  cublas_gemm<T>(handle, cublas_trans_b, cublas_trans_a, n, m, k, &alpha_val, b, ldb, a, lda,
                 &beta_val, c, ldc);
}

template<>
void Gemm(DeviceCtx* ctx, const enum CBLAS_ORDER order, enum CBLAS_TRANSPOSE trans_a,
          enum CBLAS_TRANSPOSE trans_b, const int m, const int n, const int k, const double alpha,
          const half* a, const half* b, const double beta, half* c) {
  const float alpha_f = static_cast<float>(alpha);
  const float beta_f = static_cast<float>(beta);
  int lda, ldb, ldc;
  hipblasOperation_t cublas_trans_a, cublas_trans_b;
  std::tie(lda, ldb, ldc, cublas_trans_a, cublas_trans_b) =
      PrepareToCallCublasGemm(trans_a, trans_b, m, n, k);
  if (GetCudaSmVersion() >= 500) {
    OF_CUBLAS_CHECK(hipblasGemmEx(ctx->cublas_tensor_op_math_handle(), cublas_trans_b,
                                 cublas_trans_a, n, m, k, &alpha_f, b, HIP_R_16F, ldb, a,
                                 HIP_R_16F, lda, &beta_f, c, HIP_R_16F, ldc, HIP_R_32F,
                                 CUBLAS_GEMM_DFALT_TENSOR_OP));
  } else {
    OF_CUBLAS_CHECK(cublasSgemmEx(ctx->cublas_tensor_op_math_handle(), cublas_trans_b,
                                  cublas_trans_a, n, m, k, &alpha_f, b, HIP_R_16F, ldb, a,
                                  HIP_R_16F, lda, &beta_f, c, HIP_R_16F, ldc));
  }
}

std::tuple<int, int, int> CalcMNKForGemm(enum CBLAS_TRANSPOSE trans_a, const Blob* a,
                                         const Blob* c) {
  const auto& a_shape = a->shape_view();
  const auto& c_shape = c->shape_view();
  int m = c_shape.At(0);
  int n = c_shape.Count(1);
  int k = (trans_a == CblasNoTrans) ? a_shape.Count(1) : a_shape.At(0);
  return std::make_tuple(m, n, k);
}

std::tuple<int, int, int, int, int, int, hipblasOperation_t, hipblasOperation_t>
PrepareToCallBatchedGemm(const enum CBLAS_TRANSPOSE trans_a, const enum CBLAS_TRANSPOSE trans_b,
                         int batch_size, int m, int n, int k) {
  const int a_stride = m * k;
  const int b_stride = k * n;
  const int c_stride = m * n;
  const int lda = (trans_a == CblasNoTrans) ? k : m;
  const int ldb = (trans_b == CblasNoTrans) ? n : k;
  const int ldc = n;
  hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
  hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
  return std::make_tuple(a_stride, b_stride, c_stride, lda, ldb, ldc, cublas_trans_a,
                         cublas_trans_b);
}

template<typename T>
hipDataType GetCudaDataType4BatchedGemm() {
  return CudaDataType<T>::value;
}

template<>
hipDataType GetCudaDataType4BatchedGemm<half>() {
  return HIP_R_16F;
}

template<typename T>
void BatchedGemmImpl(DeviceCtx* ctx, const enum CBLAS_ORDER order,
                     const enum CBLAS_TRANSPOSE trans_a, const enum CBLAS_TRANSPOSE trans_b,
                     int batch_size, int m, int n, int k, const double alpha, const T* a,
                     const T* b, const double beta, T* c) {
  int a_stride, b_stride, c_stride;
  int lda, ldb, ldc;
  const T alpha_val = static_cast<T>(alpha);
  const T beta_val = static_cast<T>(beta);
  hipblasOperation_t cublas_trans_a, cublas_trans_b;
  std::tie(a_stride, b_stride, c_stride, lda, ldb, ldc, cublas_trans_a, cublas_trans_b) =
      PrepareToCallBatchedGemm(trans_a, trans_b, batch_size, m, n, k);

  if (CUDA_VERSION >= 9010 && GetCudaSmVersion() >= 500) {
#if CUDA_VERSION >= 9010
    hipDataType data_type = GetCudaDataType4BatchedGemm<T>();
    OF_CUBLAS_CHECK(hipblasGemmStridedBatchedEx(
        ctx->cublas_pmh_handle(), cublas_trans_b, cublas_trans_a, n, m, k,
        reinterpret_cast<const void*>(&alpha_val), reinterpret_cast<const void*>(b), data_type, ldb,
        b_stride, reinterpret_cast<const void*>(a), data_type, lda, a_stride,
        reinterpret_cast<const void*>(&beta_val), reinterpret_cast<void*>(c), data_type, ldc,
        c_stride, batch_size, data_type, HIPBLAS_GEMM_DEFAULT));
#endif
  } else {
    cublas_gemmStridedBatched<T>(ctx->cublas_pmh_handle(), cublas_trans_b, cublas_trans_a, n, m, k,
                                 &alpha_val, b, ldb, b_stride, a, lda, a_stride, &beta_val, c, ldc,
                                 c_stride, batch_size);
  }
}

#if CUDA_VERSION >= 9010
template<>
void BatchedGemmImpl(DeviceCtx* ctx, const enum CBLAS_ORDER order,
                     const enum CBLAS_TRANSPOSE trans_a, const enum CBLAS_TRANSPOSE trans_b,
                     int batch_size, int m, int n, int k, const double alpha, const half* a,
                     const half* b, const double beta, half* c) {
  int a_stride, b_stride, c_stride;
  int lda, ldb, ldc;
  hipblasOperation_t cublas_trans_a, cublas_trans_b;
  std::tie(a_stride, b_stride, c_stride, lda, ldb, ldc, cublas_trans_a, cublas_trans_b) =
      PrepareToCallBatchedGemm(trans_a, trans_b, batch_size, m, n, k);

  if (GetCudaSmVersion() >= 500) {
    const float alpha_f = static_cast<float>(alpha);
    const float beta_f = static_cast<float>(beta);
#if CUDA_VERSION >= 11000
    hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;
#else
    hipblasGemmAlgo_t algo = CUBLAS_GEMM_DFALT_TENSOR_OP;
#endif
    OF_CUBLAS_CHECK(hipblasGemmStridedBatchedEx(
        ctx->cublas_tensor_op_math_handle(), cublas_trans_b, cublas_trans_a, n, m, k, &alpha_f,
        reinterpret_cast<const void*>(b), HIP_R_16F, ldb, b_stride,
        reinterpret_cast<const void*>(a), HIP_R_16F, lda, a_stride, &beta_f,
        reinterpret_cast<void*>(c), HIP_R_16F, ldc, c_stride, batch_size, HIP_R_32F, algo));
  } else {
    const half alpha_h = static_cast<half>(alpha);
    const half beta_h = static_cast<half>(beta);
    cublas_gemmStridedBatched<half>(ctx->cublas_tensor_op_math_handle(), cublas_trans_b,
                                    cublas_trans_a, n, m, k, &alpha_h, b, ldb, b_stride, a, lda,
                                    a_stride, &beta_h, c, ldc, c_stride, batch_size);
  }
}
#endif

__global__ void AxpyHalfGpu(const int n, const half alpha, const half* x, const int incx, half* y,
                            const int incy) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) { y[i * incy] = __hfma(alpha, x[i * incx], y[i * incy]); }
#else
  HALF_CHECK_FAILED;
#endif  // __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
}

}  // namespace

void BlasIf<DeviceType::kGPU>::OFGemm(DeviceCtx* ctx, enum CBLAS_TRANSPOSE trans_a,
                                      enum CBLAS_TRANSPOSE trans_b, const int m, const int n,
                                      const int k, const double alpha, const float* a,
                                      const float* b, const double beta, float* c) {
  Gemm<float>(ctx, CblasRowMajor, trans_a, trans_b, m, n, k, alpha, a, b, beta, c);
}
void BlasIf<DeviceType::kGPU>::OFGemm(DeviceCtx* ctx, enum CBLAS_TRANSPOSE trans_a,
                                      enum CBLAS_TRANSPOSE trans_b, const int m, const int n,
                                      const int k, const double alpha, const double* a,
                                      const double* b, const double beta, double* c) {
  Gemm<double>(ctx, CblasRowMajor, trans_a, trans_b, m, n, k, alpha, a, b, beta, c);
}
void BlasIf<DeviceType::kGPU>::OFGemm(DeviceCtx* ctx, enum CBLAS_TRANSPOSE trans_a,
                                      enum CBLAS_TRANSPOSE trans_b, const int m, const int n,
                                      const int k, const double alpha, const float16* a,
                                      const float16* b, const double beta, float16* c) {
  Gemm<half>(ctx, CblasRowMajor, trans_a, trans_b, m, n, k, alpha, reinterpret_cast<const half*>(a),
             reinterpret_cast<const half*>(b), beta, reinterpret_cast<half*>(c));
}

void BlasIf<DeviceType::kGPU>::OFBatchedGemm(DeviceCtx* ctx, enum CBLAS_TRANSPOSE trans_a,
                                             enum CBLAS_TRANSPOSE trans_b, const int batch_size,
                                             const int m, const int n, const int k,
                                             const double alpha, const float* a, const float* b,
                                             const double beta, float* c) {
  BatchedGemmImpl<float>(ctx, CblasRowMajor, trans_a, trans_b, batch_size, m, n, k, alpha, a, b,
                         beta, c);
}
void BlasIf<DeviceType::kGPU>::OFBatchedGemm(DeviceCtx* ctx, enum CBLAS_TRANSPOSE trans_a,
                                             enum CBLAS_TRANSPOSE trans_b, const int batch_size,
                                             const int m, const int n, const int k,
                                             const double alpha, const double* a, const double* b,
                                             const double beta, double* c) {
  BatchedGemmImpl<double>(ctx, CblasRowMajor, trans_a, trans_b, batch_size, m, n, k, alpha, a, b,
                          beta, c);
}
void BlasIf<DeviceType::kGPU>::OFBatchedGemm(DeviceCtx* ctx, enum CBLAS_TRANSPOSE trans_a,
                                             enum CBLAS_TRANSPOSE trans_b, const int batch_size,
                                             const int m, const int n, const int k,
                                             const double alpha, const float16* a, const float16* b,
                                             const double beta, float16* c) {
  BatchedGemmImpl<half>(ctx, CblasRowMajor, trans_a, trans_b, batch_size, m, n, k, alpha,
                        reinterpret_cast<const half*>(a), reinterpret_cast<const half*>(b), beta,
                        reinterpret_cast<half*>(c));
}

void BlasIf<DeviceType::kGPU>::Axpy(DeviceCtx* ctx, const int n, const float alpha, const float* x,
                                    const int incx, float* y, const int incy) {
  cublas_axpy<float>(ctx->cublas_pmh_handle(), n, &alpha, x, incx, y, incy);
}

void BlasIf<DeviceType::kGPU>::Axpy(DeviceCtx* ctx, const int n, const double alpha,
                                    const double* x, const int incx, double* y, const int incy) {
  cublas_axpy<double>(ctx->cublas_pmh_handle(), n, &alpha, x, incx, y, incy);
}

void BlasIf<DeviceType::kGPU>::Axpy(DeviceCtx* ctx, const int n, const float16 alpha,
                                    const float16* x, const int incx, float16* y, const int incy) {
  AxpyHalfGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, float16_2half(alpha), reinterpret_cast<const half*>(x), incx, reinterpret_cast<half*>(y),
      incy);
}

}  // namespace oneflow
