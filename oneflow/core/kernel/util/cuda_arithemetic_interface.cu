#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/util/cuda_arithemetic_interface.h"
#include "oneflow/core/common/switch_func.h"
#include "oneflow/core/kernel/util/host_arithemetic_interface.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/kernel/util/cuda_half_util.h"

namespace oneflow {

namespace {

template<int32_t NDIMS>
struct Int32Array {
  int32_t val[NDIMS];
};

template<int32_t NDIMS>
__device__ int32_t GetXIndex(const int32_t* y_shape, const int32_t* x_strides, int32_t y_idx) {
  int32_t x_idx = 0;
  for (int32_t i = NDIMS - 1; i >= 0; --i) {
    x_idx += (y_idx % y_shape[i]) * x_strides[i];
    y_idx /= y_shape[i];
  }
  return x_idx;
}

template<int32_t NDIMS, typename T>
__global__ void TransposeGpu(const Int32Array<NDIMS> y_shape, const Int32Array<NDIMS> x_strides,
                             const int32_t elem_cnt, const T* x, T* y) {
  __shared__ int32_t x_strides_shared[NDIMS];
  __shared__ int32_t y_dims_shared[NDIMS];
  const int32_t tid = threadIdx.x;
  if (tid < NDIMS) {
    y_dims_shared[tid] = y_shape.val[tid];
    x_strides_shared[tid] = x_strides.val[tid];
  }
  __syncthreads();
  CUDA_1D_KERNEL_LOOP(y_idx, elem_cnt) {
    const int32_t x_idx = GetXIndex<NDIMS>(y_dims_shared, x_strides_shared, y_idx);
#if __CUDA_ARCH__ >= 350
    y[y_idx] = __ldg(x + x_idx);
#else
    y[y_idx] = x[x_idx];
#endif
  }
}

template<int32_t NDIMS, typename T>
void TransposeImpl(DeviceCtx* ctx, const Shape& x_shape, const Shape& y_shape,
                   const PbRf<int32_t>& permutation, const int64_t elem_cnt, const T* x, T* y) {
  CHECK_LE(y_shape.elem_cnt(), GetMaxVal<int32_t>());
  Int32Array<NDIMS> y_shape_struct;
  FOR_RANGE(int32_t, i, 0, NDIMS) { y_shape_struct.val[i] = y_shape.At(i); }
  Int32Array<NDIMS> x_strides;
  int32_t buff[NDIMS];
  int32_t cur_stride = 1;
  for (int32_t i = NDIMS - 1; i >= 0; --i) {
    buff[i] = cur_stride;
    cur_stride *= x_shape.At(i);
  }
  for (int32_t i = 0; i < NDIMS; ++i) { x_strides.val[i] = buff[permutation[i]]; }
  TransposeGpu<NDIMS, T>
      <<<SMBlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          y_shape_struct, x_strides, elem_cnt, x, y);
}

template<typename T>
struct TransposeUtil final {
#define MAKE_TRANSPOSE_SWITCH_ENTRY(func_name, NDIMS) func_name<NDIMS, T>
  DEFINE_STATIC_SWITCH_FUNC(void, TransposeImpl, MAKE_TRANSPOSE_SWITCH_ENTRY,
                            MAKE_NDIM_CTRV_SEQ(DIM_SEQ));
};

}  // namespace

#define TRANSPOSE_CHECK                               \
  CHECK_LE(y_shape.elem_cnt(), GetMaxVal<int32_t>()); \
  CHECK_EQ(num_axis, y_shape.NumAxes());              \
  CHECK_EQ(num_axis, x_shape.NumAxes())

void ArithemeticIf<DeviceType::kGPU>::Transpose(DeviceCtx* ctx, const int32_t num_axis,
                                                const Shape& x_shape, const Shape& y_shape,
                                                const PbRf<int32_t>& permutation,
                                                const int64_t elem_cnt, const float* x, float* y) {
  TRANSPOSE_CHECK;
  TransposeUtil<float>::SwitchTransposeImpl(SwitchCase(num_axis), ctx, x_shape, y_shape,
                                            permutation, elem_cnt, x, y);
}

void ArithemeticIf<DeviceType::kGPU>::Transpose(DeviceCtx* ctx, const int32_t num_axis,
                                                const Shape& x_shape, const Shape& y_shape,
                                                const PbRf<int32_t>& permutation,
                                                const int64_t elem_cnt, const double* x,
                                                double* y) {
  TRANSPOSE_CHECK;
  TransposeUtil<double>::SwitchTransposeImpl(SwitchCase(num_axis), ctx, x_shape, y_shape,
                                             permutation, elem_cnt, x, y);
}

void ArithemeticIf<DeviceType::kGPU>::Transpose(DeviceCtx* ctx, const int32_t num_axis,
                                                const Shape& x_shape, const Shape& y_shape,
                                                const PbRf<int32_t>& permutation,
                                                const int64_t elem_cnt, const float16* x,
                                                float16* y) {
  TRANSPOSE_CHECK;
  TransposeUtil<half>::SwitchTransposeImpl(SwitchCase(num_axis), ctx, x_shape, y_shape, permutation,
                                           elem_cnt, reinterpret_cast<const half*>(x),
                                           reinterpret_cast<half*>(y));
}

#undef TRANSPOSE_CHECK

// create temporary host blob store initializer result
#define BEFORE_CPU_INITIALIZE()                                     \
  RtBlobDesc blob_desc(blob->blob_desc().blob_desc_proto());        \
  char* host_raw_dptr = nullptr;                                    \
  CudaCheck(hipHostMalloc(&host_raw_dptr, blob->TotalByteSize())); \
  std::unique_ptr<Blob> host_blob;                                  \
  host_blob.reset(new Blob(nullptr, &blob_desc, host_raw_dptr));

// asynchronous copy to device
#define AFTER_CPU_INITIALIZE()                                                          \
  Memcpy<DeviceType::kGPU>(ctx, blob->mut_dptr(), host_blob->dptr(),                    \
                           blob->ByteSizeOfDataContentField(), hipMemcpyHostToDevice); \
  CudaCheck(hipStreamSynchronize(ctx->hip_stream()));                                 \
  CudaCheck(hipHostFree(host_raw_dptr));

void ArithemeticIf<DeviceType::kGPU>::InitializeWithConstConf(
    DeviceCtx* ctx, const ConstantInitializerConf& initializer_conf, Blob* blob) {
  BEFORE_CPU_INITIALIZE();
  // synchronous initialize the host blob
  ArithemeticIf<DeviceType::kCPU>::InitializeWithConstConf(nullptr, initializer_conf,
                                                           host_blob.get());
  AFTER_CPU_INITIALIZE();
}

#undef BEFORE_CPU_INITIALIZE
#undef AFTER_CPU_INITIALIZE

namespace {

template<typename T>
__global__ void MulByScalarGpu(const int64_t n, const T* x, const T y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y; }
}

template<>
__global__ void MulByScalarGpu<half>(const int64_t n, const half* x, const half y, half* z) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = __hmul(x[i], y); }
#else
  HALF_CHECK_FAILED;
#endif  // __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
}

}  // namespace

#define MUL_BY_SCALAR(T)                                                                           \
  void ArithemeticIf<DeviceType::kGPU>::MulByScalar(DeviceCtx* ctx, const int64_t n, const T* x,   \
                                                    const T y, T* z) {                             \
    MulByScalarGpu<T>                                                                              \
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z); \
  }

MUL_BY_SCALAR(float)
MUL_BY_SCALAR(double)
MUL_BY_SCALAR(int32_t)
MUL_BY_SCALAR(int64_t)

#undef MUL_BY_SCALAR

void ArithemeticIf<DeviceType::kGPU>::MulByScalar(DeviceCtx* ctx, const int64_t n, const float16* x,
                                                  const float16 y, float16* z) {
  MulByScalarGpu<half><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, reinterpret_cast<const half*>(x), float16_2half(y), reinterpret_cast<half*>(z));
}

namespace {

template<typename T>
__global__ void AddByScalarGpu(const int64_t n, const T* x, const T y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] + y; }
}

template<>
__global__ void AddByScalarGpu<half>(const int64_t n, const half* x, const half y, half* z) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = __hadd(x[i], y); }
#else
  HALF_CHECK_FAILED;
#endif  // __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
}

}  // namespace

#define ADD_BY_SCALAR(T)                                                                           \
  void ArithemeticIf<DeviceType::kGPU>::AddByScalar(DeviceCtx* ctx, const int64_t n, const T* x,   \
                                                    const T y, T* z) {                             \
    AddByScalarGpu<T>                                                                              \
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z); \
  }

ADD_BY_SCALAR(float)
ADD_BY_SCALAR(double)
ADD_BY_SCALAR(int32_t)
ADD_BY_SCALAR(int64_t)

#undef ADD_BY_SCALAR

void ArithemeticIf<DeviceType::kGPU>::AddByScalar(DeviceCtx* ctx, const int64_t n, const float16* x,
                                                  const float16 y, float16* z) {
  AddByScalarGpu<half><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, reinterpret_cast<const half*>(x), float16_2half(y), reinterpret_cast<half*>(z));
}

namespace {

template<typename T>
__global__ void MulGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y[i]; }
}

template<>
__global__ void MulGpu<half>(const int64_t n, const half* x, const half* y, half* z) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = __hmul(x[i], y[i]); }
#else
  HALF_CHECK_FAILED;
#endif  // __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
}

}  // namespace

#define MUL(T)                                                                                     \
  void ArithemeticIf<DeviceType::kGPU>::Mul(DeviceCtx* ctx, const int64_t n, const T* x,           \
                                            const T* y, T* z) {                                    \
    MulGpu<T>                                                                                      \
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z); \
  }

MUL(float)
MUL(double)

#undef MUL

void ArithemeticIf<DeviceType::kGPU>::Mul(DeviceCtx* ctx, const int64_t n, const float16* x,
                                          const float16* y, float16* z) {
  MulGpu<half><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, reinterpret_cast<const half*>(x), reinterpret_cast<const half*>(y),
      reinterpret_cast<half*>(z));
}

namespace {

template<typename T>
__global__ void MulByGpuScalarGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y[0]; }
}

template<>
__global__ void MulByGpuScalarGpu<half>(const int64_t n, const half* x, const half* y, half* z) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = __hmul(x[i], y[0]); }
#else
  HALF_CHECK_FAILED;
#endif  // __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
}

}  // namespace

#define MUL_BY_GPU_SCALAR(T)                                                                       \
  void ArithemeticIf<DeviceType::kGPU>::MulByGpuScalar(DeviceCtx* ctx, const int64_t n,            \
                                                       const T* x, const T* y, T* z) {             \
    MulByGpuScalarGpu<T>                                                                           \
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z); \
  }

MUL_BY_GPU_SCALAR(float)
MUL_BY_GPU_SCALAR(double)
MUL_BY_GPU_SCALAR(int32_t)
MUL_BY_GPU_SCALAR(int64_t)

#undef MUL_BY_GPU_SCALAR

void ArithemeticIf<DeviceType::kGPU>::MulByGpuScalar(DeviceCtx* ctx, const int64_t n,
                                                     const float16* x, const float16* y,
                                                     float16* z) {
  MulByGpuScalarGpu<half>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          n, reinterpret_cast<const half*>(x), reinterpret_cast<const half*>(y),
          reinterpret_cast<half*>(z));
}

namespace {

template<typename T>
__global__ void PowByIntScalarGpu(const int64_t n, const T* x, const int32_t y, T* z);

template<>
__global__ void PowByIntScalarGpu<float>(const int64_t n, const float* x, const int32_t y,
                                         float* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = powf(x[i], y); }
}

template<>
__global__ void PowByIntScalarGpu<double>(const int64_t n, const double* x, const int32_t y,
                                          double* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = pow(x[i], y); }
}

template<>
__global__ void PowByIntScalarGpu<int32_t>(const int64_t n, const int32_t* x, const int32_t y,
                                           int32_t* z) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    z[i] = static_cast<int32_t>(powf(static_cast<float>(x[i]), static_cast<float>(y)));
  }
}

template<>
__global__ void PowByIntScalarGpu<int64_t>(const int64_t n, const int64_t* x, const int32_t y,
                                           int64_t* z) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    z[i] = static_cast<int64_t>(pow(static_cast<double>(x[i]), static_cast<double>(y)));
  }
}

}  // namespace

#define POW_BY_INT_SCALAR(T)                                                                       \
  void ArithemeticIf<DeviceType::kGPU>::PowByIntScalar(DeviceCtx* ctx, const int64_t n,            \
                                                       const T* x, const int32_t y, T* z) {        \
    PowByIntScalarGpu<T>                                                                           \
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z); \
  }

POW_BY_INT_SCALAR(float)
POW_BY_INT_SCALAR(double)
POW_BY_INT_SCALAR(int32_t)
POW_BY_INT_SCALAR(int64_t)

#undef POW_BY_INT_SCALAR

}  // namespace oneflow
