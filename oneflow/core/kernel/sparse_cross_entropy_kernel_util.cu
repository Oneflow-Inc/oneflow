#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/sparse_cross_entropy_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T, typename K>
__global__ void ComputeEntropyGpu(int64_t num_instances, int64_t num_classes, const T* x,
                                  const K* labels, T* y) {
  CUDA_1D_KERNEL_LOOP(i, num_instances) {
    K label = labels[i];
    assert(label >= 0);
    assert(label < num_classes);
    y[i] = -SafeLog(x[i * num_classes + label]);
  }
}

template<typename T, typename K>
__global__ void ComputeDiffGpu(int64_t num_instances, int64_t num_classes, const T* x,
                               const K* labels, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, num_instances) {
    K label = labels[i];
    assert(label >= 0);
    assert(label < num_classes);
    dx[i * num_classes + label] = -1 / MaxWithLogThreshold(x[i * num_classes + label]);
  }
}

template<typename T, typename K>
__global__ void ComputeDiffGpu(int64_t num_instances, int64_t num_classes, const T* x,
                               const K* labels, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, num_instances) {
    K label = labels[i];
    assert(label >= 0);
    assert(label < num_classes);
    dx[i * num_classes + label] = -dy[i] / MaxWithLogThreshold(x[i * num_classes + label]);
  }
}

}  // namespace

template<typename T, typename K>
struct SparseCrossEntropyKernelUtil<DeviceType::kGPU, T, K> {
  static void ComputeEntropy(DeviceCtx* ctx, int64_t num_instances, int64_t num_classes, const T* x,
                             const K* labels, T* y) {
    ComputeEntropyGpu<<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0,
                        ctx->hip_stream()>>>(num_instances, num_classes, x, labels, y);
  }

  static void ComputeDiff(DeviceCtx* ctx, int64_t num_instances, int64_t num_classes, const T* x,
                          const K* labels, T* dx) {
    ComputeDiffGpu<<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0,
                     ctx->hip_stream()>>>(num_instances, num_classes, x, labels, dx);
  }

  static void ComputeDiff(DeviceCtx* ctx, int64_t num_instances, int64_t num_classes, const T* x,
                          const K* labels, const T* dy, T* dx) {
    ComputeDiffGpu<<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0,
                     ctx->hip_stream()>>>(num_instances, num_classes, x, labels, dy, dx);
  }
};

#define INSTANTIATE_SPARSE_CROSS_ENTROPY_KERNEL_UTIL_GPU(data_type_pair, index_type_pair)          \
  template struct SparseCrossEntropyKernelUtil<DeviceType::kGPU, OF_PP_PAIR_FIRST(data_type_pair), \
                                               OF_PP_PAIR_FIRST(index_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_SPARSE_CROSS_ENTROPY_KERNEL_UTIL_GPU,
                                 FLOATING_DATA_TYPE_SEQ, INT_DATA_TYPE_SEQ);
#undef INSTANTIATE_SPARSE_CROSS_ENTROPY_KERNEL_UTIL_GPU

}  // namespace oneflow
