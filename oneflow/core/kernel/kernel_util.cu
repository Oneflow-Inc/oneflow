#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <hipcub/hipcub.hpp>
#include <math.h>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/new_kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void MulGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y[i]; }
}

template<typename T>
__global__ void SqrtGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::sqrt(x[i]); }
}

template<typename T>
__global__ void AxpyGpu(const int n, const T alpha, const T* x, const int incx, T* y,
                        const int incy) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i * incy] += alpha * x[i * incx]; }
}

template<typename T>
__global__ void gpu_assign_add(const int64_t n, T* out, const T* in_1) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    if (in_1[i]) { out[i] += in_1[i]; }
  }
}

template<typename T>
__global__ void gpu_assign_add(const int64_t n, T* out, const T* in_1, const T* in_2) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] += in_1[i] + in_2[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i]; }
}
template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i];
  }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6, const T* in_7) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i] + in_7[i];
  }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6, const T* in_7,
                        const T* in_8) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] =
        in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i] + in_7[i] + in_8[i];
  }
}

template<typename T>
__global__ void gpu_set(const T value, T* addr) {
  *addr = value;
}

hipblasOperation_t CblasTrans2CublasTrans(CBLAS_TRANSPOSE trans) {
  hipblasOperation_t cublas_trans;
  if (trans == CBLAS_TRANSPOSE::CblasNoTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_N;
  } else if (trans == CBLAS_TRANSPOSE::CblasTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_T;
  } else if (trans == CBLAS_TRANSPOSE::CblasConjTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_C;
  } else {
    // do nothing
  }
  return cublas_trans;
}

template<int32_t NDIMS>
struct Int32Array {
  int32_t val[NDIMS];
};

template<typename T>
__global__ void CopyColsRegionGpu(const int64_t row_num, const int64_t col_num, const T* x,
                                  const int64_t x_col_offset, const int64_t x_lda, T* y,
                                  const int64_t y_col_offset, const int64_t y_lda) {
  CUDA_1D_KERNEL_LOOP(index, row_num * col_num) {
    const int64_t i = index / col_num;
    const int64_t j = index % col_num;
    y[i * y_lda + y_col_offset + j] = x[i * x_lda + x_col_offset + j];
  }
}

template<int32_t NDIMS>
__device__ int32_t GetXIndex(const int32_t* y_shape, const int32_t* x_strides, int32_t y_idx) {
  int32_t x_idx = 0;
  for (int32_t i = NDIMS - 1; i >= 0; --i) {
    x_idx += (y_idx % y_shape[i]) * x_strides[i];
    y_idx /= y_shape[i];
  }
  return x_idx;
}

template<int32_t NDIMS, typename T>
__global__ void TransposeGpu(const Int32Array<NDIMS> y_shape, const Int32Array<NDIMS> x_strides,
                             const int32_t elem_cnt, const T* x, T* y) {
  __shared__ int32_t x_strides_shared[NDIMS];
  __shared__ int32_t y_dims_shared[NDIMS];
  const int32_t tid = threadIdx.x;
  if (tid < NDIMS) {
    y_dims_shared[tid] = y_shape.val[tid];
    x_strides_shared[tid] = x_strides.val[tid];
  }
  __syncthreads();
  CUDA_1D_KERNEL_LOOP(y_idx, elem_cnt) {
    const int32_t x_idx = GetXIndex<NDIMS>(y_dims_shared, x_strides_shared, y_idx);
#if __CUDA_ARCH__ >= 350
    y[y_idx] = __ldg(x + x_idx);
#else
    y[y_idx] = x[x_idx];
#endif
  }
}

template<int32_t NDIMS, typename T>
void Transpose(DeviceCtx* ctx, const ShapeView& x_shape, const ShapeView& y_shape,
               const PbRf<int32_t>& permutation, const int64_t elem_cnt, const T* x, T* y) {
  CHECK_LE(y_shape.elem_cnt(), GetMaxVal<int32_t>());
  Int32Array<NDIMS> y_shape_struct;
  FOR_RANGE(int32_t, i, 0, NDIMS) { y_shape_struct.val[i] = y_shape.At(i); }
  Int32Array<NDIMS> x_strides;
  int32_t buff[NDIMS];
  int32_t cur_stride = 1;
  for (int32_t i = NDIMS - 1; i >= 0; --i) {
    buff[i] = cur_stride;
    cur_stride *= x_shape.At(i);
  }
  for (int32_t i = 0; i < NDIMS; ++i) { x_strides.val[i] = buff[permutation[i]]; }
  TransposeGpu<NDIMS, T>
      <<<SMBlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          y_shape_struct, x_strides, elem_cnt, x, y);
}

template<typename T>
struct TransposeUtil final {
#define MAKE_TRANSPOSE_SWITCH_ENTRY(func_name, NDIMS) func_name<NDIMS, T>
  DEFINE_STATIC_SWITCH_FUNC(void, Transpose, MAKE_TRANSPOSE_SWITCH_ENTRY,
                            MAKE_NDIM_CTRV_SEQ(DIM_SEQ))
};

template<typename T>
__global__ void AssignStridedAddrGpu(T** dev_ptrs, T* start_ptr, int32_t stride_len,
                                     int32_t stride_num) {
  CUDA_1D_KERNEL_LOOP(i, stride_num) { dev_ptrs[i] = start_ptr + i * stride_len; }
}

template<typename T>
void AssignStridedAddr(DeviceCtx* ctx, T** dev_ptrs, T* start_ptr, int stride_len, int stride_num) {
  AssignStridedAddrGpu<T>
      <<<BlocksNum4ThreadsNum(stride_num), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          dev_ptrs, start_ptr, stride_len, stride_num);
}

}  // namespace

#define MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY(func_name, T) hipcub::DeviceReduce::func_name<T*, T*>
DEFINE_STATIC_SWITCH_FUNC(hipError_t, Sum, MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY,
                          MAKE_DATA_TYPE_CTRV_SEQ(FLOATING_DATA_TYPE_SEQ));

#undef MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY

#define KU_IF_METHOD                     \
  template<typename T, typename Derived> \
  void GpuKernelUtilIf<T, Derived>::

KU_IF_METHOD CopyColsRegion(DeviceCtx* ctx, const int64_t row_num, const int64_t col_num,
                            const T* x, const int64_t x_col_offset, const int64_t x_lda, T* y,
                            const int64_t y_col_offset, const int64_t y_lda) {
  CopyColsRegionGpu<T>
      <<<BlocksNum4ThreadsNum(row_num * col_num), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          row_num, col_num, x, x_col_offset, x_lda, y, y_col_offset, y_lda);
}

KU_IF_METHOD Transpose(DeviceCtx* ctx, const int32_t num_axis, const ShapeView& x_shape,
                       const ShapeView& y_shape, const PbRf<int32_t>& permutation,
                       const int64_t elem_cnt, const T* x, T* y) {
  CHECK_LE(y_shape.elem_cnt(), GetMaxVal<int32_t>());
  CHECK_EQ(num_axis, y_shape.NumAxes());
  CHECK_EQ(num_axis, x_shape.NumAxes());
  TransposeUtil<T>::SwitchTranspose(SwitchCase(num_axis), ctx, x_shape, y_shape, permutation,
                                    elem_cnt, x, y);
}

KU_IF_METHOD InitializeWithConf(DeviceCtx* ctx, const InitializerConf& initializer_conf,
                                uint32_t random_seed, Blob* blob) {
  WithHostBlobAndStreamSynchronizeEnv(ctx, blob, [&](Blob* host_blob) {
    KernelUtil<DeviceType::kCPU, T>::InitializeWithConf(nullptr, initializer_conf, random_seed,
                                                        host_blob);
  });
}
KU_IF_METHOD Set(DeviceCtx* ctx, const T value, T* addr) {
  gpu_set<T><<<1, 1, 0, ctx->hip_stream()>>>(value, addr);
}

#define KU_FLOATING_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kGPU, T, typename std::enable_if<IsFloating<T>::value>::type>::

KU_FLOATING_METHOD Dot(DeviceCtx* ctx, const int n, const T* x, const int incx, const T* y,
                       const int incy, T* result) {
  cublas_dot<T>(ctx->cublas_pmd_handle(), n, x, incx, y, incy, result);
}
KU_FLOATING_METHOD Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx,
                        T* y, const int incy) {
  cublas_axpy<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx, y, incy);
}
KU_FLOATING_METHOD Axpy(DeviceCtx* ctx, const int n, const T* alpha, const T* x, const int incx,
                        T* y, const int incy) {
  cublas_axpy<T>(ctx->cublas_pmd_handle(), n, alpha, x, incx, y, incy);
}

KU_FLOATING_METHOD Mul(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, T* z) {
  MulGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z);
}
KU_FLOATING_METHOD Sqrt(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
  SqrtGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0) {
  gpu_add<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, out, in_0);
}
KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1) {
  if (out == in_0) {
    gpu_assign_add<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, out, in_1);
  } else {
    gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, out, in_0, in_1);
  }
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2) {
  if (out == in_0) {
    gpu_assign_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, out, in_1, in_2);
  } else {
    gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, out, in_0, in_1, in_2);
  }
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6, const T* in_7) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6, in_7);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6, const T* in_7, const T* in_8) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6, in_7, in_8);
}

#define KU_INTEGRAL_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kGPU, T, typename std::enable_if<IsIntegral<T>::value>::type>::

KU_INTEGRAL_METHOD Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx,
                        T* y, const int incy) {
  AxpyGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, alpha, x, incx, y, incy);
}

KU_INTEGRAL_METHOD Mul(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, T* z) {
  MulGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z);
}

#define INSTANTIATE_KERNEL_UTIL(type_cpp, type_proto)                                \
  template struct GpuKernelUtilIf<type_cpp, KernelUtil<DeviceType::kGPU, type_cpp>>; \
  template struct KernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KERNEL_UTIL, ARITHMETIC_DATA_TYPE_SEQ);

template<typename T, typename U>
__global__ void CastOnGpu(const T* in, U* out, int64_t elem_num) {
  CUDA_1D_KERNEL_LOOP(i, elem_num) { out[i] = static_cast<U>(in[i]); }
}

template<>
__global__ void CastOnGpu<float, half>(const float* in, half* out, int64_t elem_num) {
  const int64_t elem_num_2 = elem_num / 2;
  const auto* in_2 = reinterpret_cast<const float2*>(in);
  auto* out_2 = reinterpret_cast<half2*>(out);
  CUDA_1D_KERNEL_LOOP(i, elem_num_2) { out_2[i] = __float22half2_rn(in_2[i]); }
  if (elem_num % 2 == 1 && blockIdx.x == 0 && threadIdx.x == 0) {
    out[elem_num - 1] = __float2half(in[elem_num - 1]);
  }
}

template<>
__global__ void CastOnGpu<half, float>(const half* in, float* out, int64_t elem_num) {
  const int64_t elem_num_2 = elem_num / 2;
  const auto* in_2 = reinterpret_cast<const half2*>(in);
  auto* out_2 = reinterpret_cast<float2*>(out);
  CUDA_1D_KERNEL_LOOP(i, elem_num_2) { out_2[i] = __half22float2(in_2[i]); }
  if (elem_num % 2 == 1 && blockIdx.x == 0 && threadIdx.x == 0) {
    out[elem_num - 1] = __half2float(in[elem_num - 1]);
  }
}

template<typename T, typename U>
void CopyElemOnGpu(DeviceCtx* ctx, const T* in_dptr, U* out_dptr, int64_t elem_num) {
  if (elem_num == 0) { return; }
  if (std::is_same<T, U>::value) {
    Memcpy<DeviceType::kGPU>(ctx, out_dptr, in_dptr, elem_num * sizeof(T));
  } else {
    CastOnGpu<T, U>
        <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            in_dptr, out_dptr, elem_num);
  }
}

template<>
void CopyElemOnGpu<float, float16>(DeviceCtx* ctx, const float* in_dptr, float16* out_dptr,
                                   int64_t elem_num) {
  if (RoundUp(elem_num, 2) == 0) { return; }
  CastOnGpu<float, half>
      <<<BlocksNum4ThreadsNum(RoundUp(elem_num, 2) / 2), kCudaThreadsNumPerBlock, 0,
         ctx->hip_stream()>>>(in_dptr, reinterpret_cast<half*>(out_dptr), elem_num);
}

template<>
void CopyElemOnGpu<float16, float>(DeviceCtx* ctx, const float16* in_dptr, float* out_dptr,
                                   int64_t elem_num) {
  if (RoundUp(elem_num, 2) == 0) { return; }
  CastOnGpu<half, float>
      <<<BlocksNum4ThreadsNum(RoundUp(elem_num, 2) / 2), kCudaThreadsNumPerBlock, 0,
         ctx->hip_stream()>>>(reinterpret_cast<const half*>(in_dptr), out_dptr, elem_num);
}

#define INSTANTIATE_COPY_ELEM_ON_GPU(T, U) \
  template void CopyElemOnGpu(DeviceCtx* ctx, const T* in_dptr, U* out_dptr, int64_t elem_num);

#define MAKE_COPY_ELEM_ON_GPU_ENTRY(TPair, UPair) \
  INSTANTIATE_COPY_ELEM_ON_GPU(OF_PP_PAIR_FIRST(TPair), OF_PP_PAIR_FIRST(UPair))

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_COPY_ELEM_ON_GPU_ENTRY, POD_DATA_TYPE_SEQ, POD_DATA_TYPE_SEQ)

}  // namespace oneflow
