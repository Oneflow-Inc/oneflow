#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void ExpGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::exp(x[i]); }
}

template<typename T>
__global__ void DivGpu(const int64_t n, T* x, const T* alpha_ptr) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = x[i] / (*alpha_ptr); }
}

template<typename T>
__global__ void MulGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y[i]; }
}

hipblasOperation_t CblasTrans2CublasTrans(CBLAS_TRANSPOSE trans) {
  hipblasOperation_t cublas_trans;
  if (trans == CBLAS_TRANSPOSE::CblasNoTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_N;
  } else if (trans == CBLAS_TRANSPOSE::CblasTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_T;
  } else if (trans == CBLAS_TRANSPOSE::CblasConjTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_C;
  } else {
    // do nothing
  }
  return cublas_trans;
}

}  // namespace

template<>
void Memcpy<DeviceType::kGPU>(DeviceCtx* ctx, void* dst, const void* src,
                              size_t sz, hipMemcpyKind kind) {
  CudaCheck(hipMemcpyAsync(dst, src, sz, kind, ctx->hip_stream()));
}

template<>
void Memset<DeviceType::kGPU>(DeviceCtx* ctx, void* dst, const char value,
                              size_t sz) {
  CudaCheck(hipMemsetAsync(dst, value, sz, ctx->hip_stream()));
}

template<typename T>
struct KernelUtil<DeviceType::kGPU, T> final {
  static void Dot(DeviceCtx* ctx, const int n, const T* x, const int incx,
                  const T* y, const int incy, T* result) {
    cublas_dot(ctx->cublas_handle(), n, x, incx, y, incy, result);
  }
  static void Copy(DeviceCtx* ctx, const int n, const T* x, const int incx,
                   T* y, const int incy) {
    cublas_copy(ctx->cublas_handle(), n, x, incx, y, incy);
  }
  static void Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x,
                   const int incx, T* y, const int incy) {
    cublas_axpy(ctx->cublas_handle(), n, &alpha, x, incx, y, incy);
  }
  static void Scal(DeviceCtx* ctx, const int n, const T alpha, T* x,
                   const int incx) {
    cublas_scal(ctx->cublas_handle(), n, &alpha, x, incx);
  }
  static void Max(DeviceCtx* ctx, const int64_t n, const T* x, T* max_ptr,
                  T* temp_storage, size_t temp_storage_bytes) {
    hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, x, max_ptr, n,
                           ctx->hip_stream());
  }
  static void Exp(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
    ExpGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                ctx->hip_stream()>>>(n, x, y);
  }
  static void Sum(DeviceCtx* ctx, const int64_t n, const T* x, T* sum_ptr,
                  T* temp_storage, size_t temp_storage_bytes) {
    hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes, x, sum_ptr, n,
                           ctx->hip_stream());
  }
  static void Div(DeviceCtx* ctx, const int64_t n, T* x, const T* alpha) {
    DivGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                ctx->hip_stream()>>>(n, x, alpha);
  }
  static void Mul(DeviceCtx* ctx, const int64_t n, const T* x, const T* y,
                  T* z) {
    MulGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                ctx->hip_stream()>>>(n, x, y, z);
  }
  static void Gemv(DeviceCtx* ctx, const enum CBLAS_TRANSPOSE trans, int m,
                   int n, const T alpha, const T* a, int lda, const T* x,
                   const int incx, const T beta, T* y, const int incy) {
    hipblasOperation_t cublas_trans = CblasTrans2CublasTrans(trans);
    cublas_gemv(ctx->cublas_handle(), cublas_trans, n, m, &alpha, a, lda, x,
                incx, &beta, y, incy);
  }
  static void Gemm(DeviceCtx* ctx, const enum CBLAS_ORDER order,
                   const enum CBLAS_TRANSPOSE trans_a,
                   const enum CBLAS_TRANSPOSE trans_b, const int m, const int n,
                   const int k, const T alpha, const T* a, const int lda,
                   const T* b, const int ldb, const T beta, T* c,
                   const int ldc) {
    hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
    hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
    cublas_gemm(ctx->cublas_handle(), cublas_trans_b, cublas_trans_a, n, m, k,
                &alpha, b, ldb, a, lda, &beta, c, ldc);
  }

  static void Initialize(DeviceCtx* ctx,
                         const InitializerConf& initializer_conf,
                         uint32_t random_seed, Blob* blob) {
    // create temporary host blob store initializer result
    BlobDesc blob_desc = BlobDesc(blob->blob_desc());
    char* host_raw_dptr = nullptr;
    CudaCheck(hipHostMalloc(&host_raw_dptr, blob->TotalByteSize()));
    Blob host_blob(nullptr, &blob_desc, host_raw_dptr);
    // synchronous initialize the host blob
    KernelUtil<DeviceType::kCPU, T>::Initialize(nullptr, initializer_conf,
                                                random_seed, &host_blob);
    // asynchronous copy to device
    Memcpy<DeviceType::kGPU>(ctx, blob->mut_dptr(), host_blob.dptr(),
                             blob->ByteSizeOfDataContentField(),
                             hipMemcpyHostToDevice);
    hipStreamSynchronize(ctx->hip_stream());
    CudaCheck(hipHostFree(host_raw_dptr));
  }

  static void InitializeWithModelDir(DeviceCtx* ctx, int32_t part_id,
                                     int32_t part_num,
                                     const std::string& model_dir, Blob* blob,
                                     const std::string& bn_in_op,
                                     int32_t dim_num, int64_t num_in_each_dim) {
    BlobDesc blob_desc = BlobDesc(blob->blob_desc());
    char* host_raw_dptr = nullptr;
    CudaCheck(hipHostMalloc(&host_raw_dptr, blob->TotalByteSize()));
    Blob host_blob(nullptr, &blob_desc, host_raw_dptr);
    KernelUtil<DeviceType::kCPU, T>::InitializeWithModelDir(
        ctx, part_id, part_num, model_dir, &host_blob, bn_in_op, dim_num,
        num_in_each_dim);

    Memcpy<DeviceType::kGPU>(ctx, blob->mut_dptr(), host_blob.dptr(),
                             blob->ByteSizeOfDataContentField(),
                             hipMemcpyHostToDevice);
    hipStreamSynchronize(ctx->hip_stream());
    CudaCheck(hipHostFree(host_raw_dptr));
  }
};

#define INSTANTIATE_KERNEL_UTIL(type_cpp, type_proto) \
  template struct KernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ);

#define DEFINE_INT_KERNEL_UTIL(T, type_proto)                                 \
  template void KernelUtil<DeviceType::kGPU, T>::Sum(                         \
      DeviceCtx* ctx, const int64_t n, const T* x, T* sum_ptr,                \
      T* temp_storage, size_t temp_storage_bytes);                            \
                                                                              \
  template<>                                                                  \
  void KernelUtil<DeviceType::kGPU, T>::Axpy(                                 \
      DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx, \
      T* y, const int incy) {                                                 \
    FOR_RANGE(int, i, 0, n) {                                                 \
      *y += alpha * *x;                                                       \
      x += incx;                                                              \
      y += incy;                                                              \
    }                                                                         \
  }

OF_PP_FOR_EACH_TUPLE(DEFINE_INT_KERNEL_UTIL, INT_DATA_TYPE_SEQ);

}  // namespace oneflow
