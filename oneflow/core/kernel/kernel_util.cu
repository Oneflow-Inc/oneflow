#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void ExpGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::exp(x[i]); }
}

template<typename T>
__global__ void DivGpu(const int64_t n, T* x, const T* alpha_ptr) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = x[i] / (*alpha_ptr); }
}

template<typename T>
__global__ void MulGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y[i]; }
}

}  // namespace

template<>
void Memcpy<DeviceType::kGPU>(DeviceCtx* ctx, void* dst, const void* src,
                              size_t sz, hipMemcpyKind kind) {
  CudaCheck(hipMemcpyAsync(dst, src, sz, kind, ctx->hip_stream()));
}

template<>
void Memset<DeviceType::kGPU>(DeviceCtx* ctx, void* dst, const char value,
                              size_t sz) {
  CudaCheck(hipMemsetAsync(dst, value, sz, ctx->hip_stream()));
}

template<typename T>
class KernelUtil<DeviceType::kGPU, T> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(KernelUtil);
  KernelUtil() = delete;

  static void BlasAxpy(DeviceCtx* ctx, const int n, const T alpha, const T* x,
                       const int incx, T* y, const int incy) {
    cublas_axpy(ctx->cublas_handle(), n, &alpha, x, incx, y, incy);
  }

  static void BlasScal(DeviceCtx* ctx, const int n, const T alpha, T* x,
                       const int incx) {
    cublas_scal(ctx->cublas_handle(), n, &alpha, x, incx);
  }

  static void Max(DeviceCtx* ctx, const int64_t n, const T* x, T* max_ptr,
                  T* temp_storage, size_t temp_storage_bytes) {
    hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, x, max_ptr, n,
                           ctx->hip_stream());
  }

  static void Exp(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
    ExpGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                ctx->hip_stream()>>>(n, x, y);
  }

  static void Sum(DeviceCtx* ctx, const int64_t n, const T* x, T* sum_ptr,
                  T* temp_storage, size_t temp_storage_bytes) {
    hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes, x, sum_ptr, n,
                           ctx->hip_stream());
  }

  static void Div(DeviceCtx* ctx, const int64_t n, T* x, const T* alpha_ptr) {
    DivGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                ctx->hip_stream()>>>(n, x, alpha_ptr);
  }

  static void Mul(DeviceCtx* ctx, const int64_t n, const T* x, const T* y,
                  T* z) {
    MulGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                ctx->hip_stream()>>>(n, x, y, z);
  }

  static void BlasGemv(DeviceCtx* ctx, const enum CBLAS_TRANSPOSE trans, int m,
                       int n, const T alpha, const T* a, int lda, const T* x,
                       const int incx, const T beta, T* y, const int incy) {
    hipblasOperation_t cublas_trans = CblasTrans2CublasTrans(trans);
    cublas_gemv(ctx->cublas_handle(), cublas_trans, n, m, &alpha, a, lda, x,
                incx, &beta, y, incy);
  }

  static void BlasGemm(DeviceCtx* ctx, const enum CBLAS_ORDER order,
                       const enum CBLAS_TRANSPOSE trans_a,
                       const enum CBLAS_TRANSPOSE trans_b, const int m,
                       const int n, const int k, const T alpha, const T* a,
                       const int lda, const T* b, const int ldb, const T beta,
                       T* c, const int ldc) {
    hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
    hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
    cublas_gemm(ctx->cublas_handle(), cublas_trans_b, cublas_trans_a, n, m, k,
                &alpha, b, ldb, a, lda, &beta, c, ldc);
  }

  static void BlasDot(DeviceCtx* ctx, const int n, const T* x, const int incx,
                      const T* y, const int incy, T* result) {
    cublas_dot(ctx->cublas_handle(), n, x, incx, y, incy, result);
  }

  static void BlasSwap(DeviceCtx* ctx, const int n, T* x, const int incx, T* y,
                       const int incy) {
    cublas_swap(ctx->cublas_handle(), n, x, incx, y, incy);
  }

  static void BlasCopy(DeviceCtx* ctx, const int n, const T* x, const int incx,
                       T* y, const int incy) {
    cublas_copy(ctx->cublas_handle(), n, x, incx, y, incy);
  }

  static void Fill(DeviceCtx* ctx, const FillConf& fill_conf,
                   uint32_t random_seed, Blob* blob) {
    // create temporary host blob store fill
    BlobDesc blob_desc = BlobDesc(blob->blob_desc());
    char* host_raw_dptr;
    size_t byte_size = blob->TotalByteSize();
    CudaCheck(hipHostMalloc(&host_raw_dptr, byte_size));
    Blob host_blob(&blob_desc, host_raw_dptr);
    // synchronous fill the host blob
    KernelUtil<DeviceType::kCPU, T>::Fill(fill_conf, random_seed, &host_blob);
    // asynchronous copy to device
    Memcpy<DeviceType::kGPU>(ctx, blob->mut_dptr(), host_blob.dptr(), byte_size,
                             hipMemcpyHostToDevice);
    hipStreamSynchronize(ctx->hip_stream());
    CudaCheck(hipHostFree(host_raw_dptr));
  }

  static void FillWithModelDir(DeviceCtx* ctx, int32_t part_id,
                               int32_t part_num, const std::string& model_dir,
                               Blob* blob, const std::string& bn_in_op,
                               int32_t dim_num, int64_t num_in_each_dim) {
    TODO();
    /*
    int64_t blob_size = blob->shape().elem_cnt() * sizeof(T);
    std::unique_ptr<PersistentInStream> in_stream =
        snapshot->GetInStream(lbn, part_id, part_num, dim_num,
                              num_in_each_dim * sizeof(T));
    // read model from disk to host_blob synchronously
    void* host_raw_dptr;
    CudaCheck(hipHostMalloc(&host_raw_dptr, blob_size));
    std::unique_ptr<void, std::function<void(void*)>> host_unique_ptr(
        host_raw_dptr, [&](void* dptr) { CudaCheck(hipHostFree(dptr)); });
    std::unique_ptr<Shape> host_blob_shape(new Shape(blob->shape()));
    std::unique_ptr<Blob> host_blob(
        new Blob(host_unique_ptr.get(), host_blob_shape.get()));
    in_stream->Read(host_blob->mut_dptr<char>(), blob_size);
    // copy to device blob
    KernelUtil<DeviceType::kGPU, T>::Memcpy(
        ctx, blob->mut_dptr(), host_blob->dptr(), blob_size,
        hipMemcpyHostToDevice);
        */
  }

 private:
  static hipblasOperation_t CblasTrans2CublasTrans(CBLAS_TRANSPOSE trans) {
    hipblasOperation_t cublas_trans;
    if (trans == CBLAS_TRANSPOSE::CblasNoTrans) {
      cublas_trans = hipblasOperation_t::HIPBLAS_OP_N;
    } else if (trans == CBLAS_TRANSPOSE::CblasTrans) {
      cublas_trans = hipblasOperation_t::HIPBLAS_OP_T;
    } else if (trans == CBLAS_TRANSPOSE::CblasConjTrans) {
      cublas_trans = hipblasOperation_t::HIPBLAS_OP_C;
    } else {
      // do nothing
    }
    return cublas_trans;
  }
};

#define INSTANTIATE_KERNEL_UTIL(type_cpp, type_proto) \
  template class KernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
