#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <hipcub/hipcub.hpp>
#include <math.h>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void RsqrtGpu(const int64_t n, T* x, const float epsilon) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = 1.0 / std::sqrt(x[i] + epsilon); }
}

template<typename T>
__global__ void RsqrtGpu(const int64_t n, const T* x, T* y, const float epsilon) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = 1.0 / std::sqrt(x[i] + epsilon); }
}

template<typename T>
__global__ void ExpGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::exp(x[i]); }
}

template<typename T>
__global__ void DivByConstParaPtrGpu(const int64_t n, T* x, const T* alpha_ptr) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = x[i] / (*alpha_ptr); }
}

template<typename T>
__global__ void DivGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] / y[i]; }
}

template<typename T>
__global__ void DivByConstParaGpu(const int64_t n, T* x, const T alpha) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = x[i] / alpha; }
}

template<typename T>
__global__ void ReplicateGpu(const int64_t n, T* y, const T* x) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = *x; }
}

template<typename T>
__global__ void MulGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y[i]; }
}

template<typename T>
__global__ void MulByScalarGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y[0]; }
}

template<typename T>
__global__ void AddByScalarGpu(const int64_t n, const T* x, const T y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] + y; }
}

template<typename T>
__global__ void MulByScalarParaGpu(const int64_t n, const T* x, const T y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y; }
}

template<typename T>
__global__ void ReciprocalGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = static_cast<T>(1.0) / x[i]; }
}

template<typename T>
__global__ void SquareGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = x[i] * x[i]; }
}

template<typename T>
__global__ void SqrtGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::sqrt(x[i]); }
}

template<typename T>
__global__ void AxpyGpu(const int n, const T alpha, const T* x, const int incx, T* y,
                        const int incy) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i * incy] += alpha * x[i * incx]; }
}

template<typename T>
__global__ void SigmoidForwardGpu(const int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = 1.0 / (1.0 + std::exp(-x[i])); }
}

template<typename T>
__global__ void SigmoidBackwardGpu(const int n, const T* y, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = dy[i] * y[i] * (1.0 - y[i]); }
}

template<typename T>
__global__ void TanHForwardGpu(const int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::tanh(x[i]); }
}

template<typename T>
__global__ void TanHBackwardGpu(const int n, const T* y, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = dy[i] * (1.0 - y[i] * y[i]); }
}

template<typename T>
__global__ void ReluForwardGpu(const int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = x[i] > 0 ? x[i] : 0; }
}

template<typename T>
__global__ void ReluBackwardGpu(const int n, const T* y, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = y[i] > 0 ? dy[i] : 0; }
}

template<typename T>
__global__ void gpu_assign_add(const int64_t n, T* out, const T* in_1) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    if (in_1[i]) { out[i] += in_1[i]; }
  }
}

template<typename T>
__global__ void gpu_assign_add(const int64_t n, T* out, const T* in_1, const T* in_2) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] += in_1[i] + in_2[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i]; }
}
template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i];
  }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6, const T* in_7) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i] + in_7[i];
  }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6, const T* in_7,
                        const T* in_8) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] =
        in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i] + in_7[i] + in_8[i];
  }
}

template<typename T>
__global__ void gpu_set(const T value, T* addr) {
  *addr = value;
}

hipblasOperation_t CblasTrans2CublasTrans(CBLAS_TRANSPOSE trans) {
  hipblasOperation_t cublas_trans;
  if (trans == CBLAS_TRANSPOSE::CblasNoTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_N;
  } else if (trans == CBLAS_TRANSPOSE::CblasTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_T;
  } else if (trans == CBLAS_TRANSPOSE::CblasConjTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_C;
  } else {
    // do nothing
  }
  return cublas_trans;
}

template<int32_t NDIMS>
struct Int32Array {
  int32_t val[NDIMS];
};

template<typename T>
__global__ void CopyColsRegionGpu(const int64_t row_num, const int64_t col_num, const T* x,
                                  const int64_t x_col_offset, const int64_t x_lda, T* y,
                                  const int64_t y_col_offset, const int64_t y_lda) {
  CUDA_1D_KERNEL_LOOP(index, row_num * col_num) {
    const int64_t i = index / col_num;
    const int64_t j = index % col_num;
    y[i * y_lda + y_col_offset + j] = x[i * x_lda + x_col_offset + j];
  }
}

template<int32_t NDIMS>
__device__ int32_t GetXIndex(const int32_t* y_shape, const int32_t* x_strides, int32_t y_idx) {
  int32_t x_idx = 0;
  for (int32_t i = NDIMS - 1; i >= 0; --i) {
    x_idx += (y_idx % y_shape[i]) * x_strides[i];
    y_idx /= y_shape[i];
  }
  return x_idx;
}

template<int32_t NDIMS, typename T>
__global__ void TransposeGpu(const Int32Array<NDIMS> y_shape, const Int32Array<NDIMS> x_strides,
                             const int32_t elem_cnt, const T* x, T* y) {
  __shared__ int32_t x_strides_shared[NDIMS];
  __shared__ int32_t y_dims_shared[NDIMS];
  const int32_t tid = threadIdx.x;
  if (tid < NDIMS) {
    y_dims_shared[tid] = y_shape.val[tid];
    x_strides_shared[tid] = x_strides.val[tid];
  }
  __syncthreads();
  CUDA_1D_KERNEL_LOOP(y_idx, elem_cnt) {
    const int32_t x_idx = GetXIndex<NDIMS>(y_dims_shared, x_strides_shared, y_idx);
#if __CUDA_ARCH__ >= 350
    y[y_idx] = __ldg(x + x_idx);
#else
    y[y_idx] = x[x_idx];
#endif
  }
}

template<int32_t NDIMS, typename T>
void Transpose(DeviceCtx* ctx, const ShapeView& x_shape, const ShapeView& y_shape,
               const PbRf<int32_t>& permutation, const int64_t elem_cnt, const T* x, T* y) {
  CHECK_LE(y_shape.elem_cnt(), GetMaxVal<int32_t>());
  Int32Array<NDIMS> y_shape_struct;
  FOR_RANGE(int32_t, i, 0, NDIMS) { y_shape_struct.val[i] = y_shape.At(i); }
  Int32Array<NDIMS> x_strides;
  int32_t buff[NDIMS];
  int32_t cur_stride = 1;
  for (int32_t i = NDIMS - 1; i >= 0; --i) {
    buff[i] = cur_stride;
    cur_stride *= x_shape.At(i);
  }
  for (int32_t i = 0; i < NDIMS; ++i) { x_strides.val[i] = buff[permutation[i]]; }
  TransposeGpu<NDIMS, T>
      <<<SMBlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          y_shape_struct, x_strides, elem_cnt, x, y);
}

template<typename T>
struct TransposeUtil final {
#define MAKE_TRANSPOSE_SWITCH_ENTRY(func_name, NDIMS) func_name<NDIMS, T>
  DEFINE_STATIC_SWITCH_FUNC(void, Transpose, MAKE_TRANSPOSE_SWITCH_ENTRY,
                            MAKE_NDIM_CTRV_SEQ(DIM_SEQ));
};

template<typename T, T (*reduce_core_func)(const T, const T)>
__device__ void MatrixShrinkCols(const size_t row_num, const size_t thread_col_num, const T* x,
                                 const size_t x_col_num, const size_t x_lda, T* y,
                                 const size_t y_col_num, const size_t y_lda) {
  const size_t thread_num = blockDim.x * gridDim.x;
  const size_t total_shrink_scale = thread_col_num / y_col_num;
  CUDA_1D_KERNEL_LOOP(index, row_num * thread_col_num) {
    const int32_t thread_col = index % thread_col_num;
    if (((index / thread_num) % total_shrink_scale) != thread_col / y_col_num) { continue; }
    const int32_t row = index / thread_col_num;
    const int32_t col = thread_col % y_col_num;
    const int32_t x_start = row * x_lda + col;
    const int32_t x_end = row * x_lda + x_col_num;
    T reduced = x[x_start];
    for (int32_t x_index = x_start + y_col_num; x_index < x_end; x_index += y_col_num) {
      reduced = reduce_core_func(reduced, x[x_index]);
    }
    y[row * y_lda + col] = reduced;
  }
}

template<typename T, T (*reduce_core_func)(const T, const T), size_t shift_size = 2>
__global__ void MatrixRowReduceGpu(const size_t row_num, const size_t col_num, const T* x, T* y,
                                   T* temp_storage, size_t temp_col_num) {
  const size_t temp_lda = temp_col_num;
  MatrixShrinkCols<T, reduce_core_func>(row_num, temp_lda, x, col_num, col_num, temp_storage,
                                        temp_col_num, temp_lda);
  __syncthreads();
  while (temp_col_num > (1 << shift_size)) {
    size_t new_temp_col_num = temp_col_num >> shift_size;
    MatrixShrinkCols<T, reduce_core_func>(row_num, temp_lda, temp_storage, temp_col_num, temp_lda,
                                          temp_storage, new_temp_col_num, temp_lda);
    temp_col_num = new_temp_col_num;
    __syncthreads();
  }
  MatrixShrinkCols<T, reduce_core_func>(row_num, temp_lda, temp_storage, temp_col_num, temp_lda, y,
                                        1, 1);
}

template<typename T, T (*reduce_core_func)(const T, const T), size_t shift_size = 2>
void MatrixRowReduce(DeviceCtx* ctx, const size_t row_num, const size_t col_num, const T* x, T* y,
                     void* temp_storage, const size_t temp_storage_bytes) {
  CHECK_NOTNULL(temp_storage);
  CHECK_GT(temp_storage_bytes / sizeof(T), row_num);
  const size_t temp_col_num_shift =
      std::floor(std::log2(std::min(temp_storage_bytes / sizeof(T) / row_num, col_num)));
  const size_t temp_col_num = std::min(static_cast<size_t>(kCudaThreadsNumPerBlock),
                                       static_cast<size_t>(1 << temp_col_num_shift));
  MatrixRowReduceGpu<T, reduce_core_func>
      <<<BlocksNum4ThreadsNum(row_num * temp_col_num), kCudaThreadsNumPerBlock, 0,
         ctx->hip_stream()>>>(row_num, col_num, x, y, static_cast<T*>(temp_storage), temp_col_num);
}

template<typename T>
__global__ void AssignStridedAddrGpu(T** dev_ptrs, T* start_ptr, int32_t stride_len,
                                     int32_t stride_num) {
  CUDA_1D_KERNEL_LOOP(i, stride_num) { dev_ptrs[i] = start_ptr + i * stride_len; }
}

template<typename T>
void AssignStridedAddr(DeviceCtx* ctx, T** dev_ptrs, T* start_ptr, int stride_len, int stride_num) {
  AssignStridedAddrGpu<T>
      <<<BlocksNum4ThreadsNum(stride_num), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          dev_ptrs, start_ptr, stride_len, stride_num);
}

}  // namespace

#define MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY(func_name, T) hipcub::DeviceReduce::func_name<T*, T*>
DEFINE_STATIC_SWITCH_FUNC(hipError_t, Sum, MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY,
                          MAKE_DATA_TYPE_CTRV_SEQ(FLOATING_DATA_TYPE_SEQ));

size_t GetTmpSizeForReduceSum(DataType data_type, int64_t sum_elem_num) {
  size_t tmp_storage_size;
  SwitchSum(SwitchCase(data_type), nullptr, tmp_storage_size, nullptr, nullptr, sum_elem_num);
  return tmp_storage_size;
}

#undef MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY

#define KU_IF_METHOD                     \
  template<typename T, typename Derived> \
  void GpuKernelUtilIf<T, Derived>::

KU_IF_METHOD Max(DeviceCtx* ctx, const int64_t n, const T* x, T* max_ptr, T* temp_storage,
                 size_t temp_storage_bytes) {
  OF_CUDA_CHECK(
      hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, x, max_ptr, n, ctx->hip_stream()));
}
KU_IF_METHOD Sum(DeviceCtx* ctx, const int64_t n, const T* x, T* sum_ptr, T* temp_storage,
                 size_t temp_storage_bytes) {
  OF_CUDA_CHECK(
      hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes, x, sum_ptr, n, ctx->hip_stream()));
}
KU_IF_METHOD CopyColsRegion(DeviceCtx* ctx, const int64_t row_num, const int64_t col_num,
                            const T* x, const int64_t x_col_offset, const int64_t x_lda, T* y,
                            const int64_t y_col_offset, const int64_t y_lda) {
  CopyColsRegionGpu<T>
      <<<BlocksNum4ThreadsNum(row_num * col_num), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          row_num, col_num, x, x_col_offset, x_lda, y, y_col_offset, y_lda);
}
KU_IF_METHOD RowMax(DeviceCtx* ctx, const int64_t row_num, const int64_t col_num, const T* x, T* y,
                    void* temp_storage, const size_t temp_storage_bytes) {
  MatrixRowReduce<T, ReduceCoreMax>(ctx, row_num, col_num, x, y, temp_storage, temp_storage_bytes);
}
KU_IF_METHOD RowSum(DeviceCtx* ctx, const int64_t row_num, const int64_t col_num, const T* x, T* y,
                    void* temp_storage, const size_t temp_storage_bytes) {
  MatrixRowReduce<T, ReduceCoreAdd>(ctx, row_num, col_num, x, y, temp_storage, temp_storage_bytes);
}

KU_IF_METHOD Transpose(DeviceCtx* ctx, const int32_t num_axis, const ShapeView& x_shape,
                       const ShapeView& y_shape, const PbRf<int32_t>& permutation,
                       const int64_t elem_cnt, const T* x, T* y) {
  CHECK_LE(y_shape.elem_cnt(), GetMaxVal<int32_t>());
  CHECK_EQ(num_axis, y_shape.NumAxes());
  CHECK_EQ(num_axis, x_shape.NumAxes());
  TransposeUtil<T>::SwitchTranspose(SwitchCase(num_axis), ctx, x_shape, y_shape, permutation,
                                    elem_cnt, x, y);
}

KU_IF_METHOD InitializeWithConf(DeviceCtx* ctx, const InitializerConf& initializer_conf,
                                uint32_t random_seed, Blob* blob) {
  WithHostBlobAndStreamSynchronizeEnv(ctx, blob, [&](Blob* host_blob) {
    KernelUtil<DeviceType::kCPU, T>::InitializeWithConf(nullptr, initializer_conf, random_seed,
                                                        host_blob);
  });
}
KU_IF_METHOD Set(DeviceCtx* ctx, const T value, T* addr) {
  gpu_set<T><<<1, 1, 0, ctx->hip_stream()>>>(value, addr);
}
KU_IF_METHOD Replicate(DeviceCtx* ctx, const int64_t n, T* y, const T* x) {
  ReplicateGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, y, x);
}
KU_IF_METHOD AddByScalar(DeviceCtx* ctx, const int64_t n, const T* x, const T y, T* z) {
  AddByScalarGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z);
}
KU_IF_METHOD MulByScalarPara(DeviceCtx* ctx, const int64_t n, const T* x, const T y, T* z) {
  MulByScalarParaGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z);
}

#define KU_FLOATING_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kGPU, T, typename std::enable_if<IsFloating<T>::value>::type>::

KU_FLOATING_METHOD Dot(DeviceCtx* ctx, const int n, const T* x, const int incx, const T* y,
                       const int incy, T* result) {
  cublas_dot<T>(ctx->cublas_pmd_handle(), n, x, incx, y, incy, result);
}
KU_FLOATING_METHOD Copy(DeviceCtx* ctx, const int n, const T* x, const int incx, T* y,
                        const int incy) {
  cublas_copy<T>(ctx->cublas_pmh_handle(), n, x, incx, y, incy);
}
KU_FLOATING_METHOD Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx,
                        T* y, const int incy) {
  cublas_axpy<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx, y, incy);
}
KU_FLOATING_METHOD Axpy(DeviceCtx* ctx, const int n, const T* alpha, const T* x, const int incx,
                        T* y, const int incy) {
  cublas_axpy<T>(ctx->cublas_pmd_handle(), n, alpha, x, incx, y, incy);
}
KU_FLOATING_METHOD Scal(DeviceCtx* ctx, const int n, const T alpha, T* x, const int incx) {
  cublas_scal<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx);
}
KU_FLOATING_METHOD Scal(DeviceCtx* ctx, const int n, const T* alpha, T* x, const int incx) {
  cublas_scal<T>(ctx->cublas_pmd_handle(), n, alpha, x, incx);
}
KU_FLOATING_METHOD Gemv(DeviceCtx* ctx, const enum CBLAS_TRANSPOSE trans, int m, int n,
                        const T alpha, const T* a, int lda, const T* x, const int incx,
                        const T beta, T* y, const int incy) {
  hipblasOperation_t cublas_trans = CblasTrans2CublasTrans(trans);
  cublas_gemv<T>(ctx->cublas_pmh_handle(), cublas_trans, n, m, &alpha, a, lda, x, incx, &beta, y,
                 incy);
}
KU_FLOATING_METHOD Gemm(DeviceCtx* ctx, const enum CBLAS_ORDER order,
                        const enum CBLAS_TRANSPOSE trans_a, const enum CBLAS_TRANSPOSE trans_b,
                        const int m, const int n, const int k, const T alpha, const T* a,
                        const int lda, const T* b, const int ldb, const T beta, T* c,
                        const int ldc) {
  hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
  hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
  cublas_gemm<T>(ctx->cublas_pmh_handle(), cublas_trans_b, cublas_trans_a, n, m, k, &alpha, b, ldb,
                 a, lda, &beta, c, ldc);
}
KU_FLOATING_METHOD BatchedGemm(DeviceCtx* ctx, const enum CBLAS_ORDER order,
                               const enum CBLAS_TRANSPOSE trans_a,
                               const enum CBLAS_TRANSPOSE trans_b, int batch_size, int m, int n,
                               int k, const T alpha, const T* a, const T* b, const T beta, T* c,
                               T** buf) {
  const int a_stride = m * k;
  const int b_stride = k * n;
  const int c_stride = m * n;
  const int lda = (trans_a == CblasNoTrans) ? k : m;
  const int ldb = (trans_b == CblasNoTrans) ? n : k;
  const int ldc = n;
  hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
  hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
  T** dev_a_ptrs = buf;
  T** dev_b_ptrs = buf + batch_size;
  T** dev_c_ptrs = buf + 2 * batch_size;
  AssignStridedAddr<T>(ctx, dev_a_ptrs, const_cast<T*>(a), a_stride, batch_size);
  AssignStridedAddr<T>(ctx, dev_b_ptrs, const_cast<T*>(b), b_stride, batch_size);
  AssignStridedAddr<T>(ctx, dev_c_ptrs, c, c_stride, batch_size);
#if CUDA_VERSION >= 9010
  hipDataType data_type = CudaDataType<T>::value;
  hipblasGemmBatchedEx(ctx->cublas_pmh_handle(), cublas_trans_b, cublas_trans_a, n, m, k,
                      reinterpret_cast<const void*>(&alpha),
                      reinterpret_cast<const void**>(const_cast<const T**>(dev_b_ptrs)), data_type,
                      ldb, reinterpret_cast<const void**>(const_cast<const T**>(dev_a_ptrs)),
                      data_type, lda, reinterpret_cast<const void*>(&beta),
                      reinterpret_cast<void**>(dev_c_ptrs), data_type, ldc, batch_size, data_type,
                      HIPBLAS_GEMM_DEFAULT);
#else
  cublas_gemmBatched<T>(ctx->cublas_pmh_handle(), cublas_trans_b, cublas_trans_a, n, m, k, &alpha,
                        const_cast<const T**>(dev_b_ptrs), ldb, const_cast<const T**>(dev_a_ptrs),
                        lda, &beta, dev_c_ptrs, ldc, batch_size);
#endif
}

KU_FLOATING_METHOD Exp(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
  ExpGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}
KU_FLOATING_METHOD Div(DeviceCtx* ctx, const int64_t n, T* x, const T* alpha) {
  DivByConstParaPtrGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, alpha);
}
KU_FLOATING_METHOD Div(DeviceCtx* ctx, const int64_t n, T* x, const T alpha) {
  DivByConstParaGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, alpha);
}
KU_FLOATING_METHOD Div(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, T* z) {
  DivGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z);
}
KU_FLOATING_METHOD Mul(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, T* z) {
  MulGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z);
}
KU_FLOATING_METHOD MulByScalar(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, T* z) {
  MulByScalarGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z);
}
KU_FLOATING_METHOD Reciprocal(DeviceCtx* ctx, const int n, const T* x, T* y) {
  ReciprocalGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}
KU_FLOATING_METHOD Square(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
  SquareGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}
KU_FLOATING_METHOD Sqrt(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
  SqrtGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}
KU_FLOATING_METHOD Rsqrt(DeviceCtx* ctx, const int64_t n, T* x, const float epsilon) {
  RsqrtGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, epsilon);
}
KU_FLOATING_METHOD Rsqrt(DeviceCtx* ctx, const int64_t n, const T* x, T* y, const float epsilon) {
  RsqrtGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y,
                                                                                           epsilon);
}

KU_FLOATING_METHOD Sigmoid(DeviceCtx* ctx, int64_t n, const T* x, T* y) {
  SigmoidForwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}

KU_FLOATING_METHOD SigmoidBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y,
                                   const T* dy, T* dx) {
  SigmoidBackwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, y, dy, dx);
}

KU_FLOATING_METHOD TanH(DeviceCtx* ctx, int64_t n, const T* x, T* y) {
  TanHForwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}

KU_FLOATING_METHOD TanHBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y,
                                const T* dy, T* dx) {
  TanHBackwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, y, dy, dx);
}

KU_FLOATING_METHOD Relu(DeviceCtx* ctx, int64_t n, const T* x, T* y) {
  ReluForwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}

KU_FLOATING_METHOD ReluBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y,
                                const T* dy, T* dx) {
  ReluBackwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, y, dy, dx);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0) {
  gpu_add<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, out, in_0);
}
KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1) {
  if (out == in_0) {
    gpu_assign_add<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, out, in_1);
  } else {
    gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, out, in_0, in_1);
  }
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2) {
  if (out == in_0) {
    gpu_assign_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, out, in_1, in_2);
  } else {
    gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, out, in_0, in_1, in_2);
  }
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6, const T* in_7) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6, in_7);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6, const T* in_7, const T* in_8) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6, in_7, in_8);
}

#define KU_INTEGRAL_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kGPU, T, typename std::enable_if<IsIntegral<T>::value>::type>::

KU_INTEGRAL_METHOD Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx,
                        T* y, const int incy) {
  AxpyGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, alpha, x, incx, y, incy);
}

KU_INTEGRAL_METHOD Mul(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, T* z) {
  MulGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z);
}

#define INSTANTIATE_KERNEL_UTIL(type_cpp, type_proto)                                \
  template struct GpuKernelUtilIf<type_cpp, KernelUtil<DeviceType::kGPU, type_cpp>>; \
  template struct KernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KERNEL_UTIL, ARITHMETIC_DATA_TYPE_SEQ);

template<>
__device__ int32_t gpu_atomic_add(int32_t* address, const int32_t val) {
  return atomicAdd(address, val);
}

template<>
__device__ float gpu_atomic_add(float* address, float val) {
  return atomicAdd(address, val);
}

template<>
__device__ half gpu_atomic_add(half* address, half val) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700 && CUDA_VERSION >= 10000
  return atomicAdd(address, val);
#else
  __trap();
#endif
}

template<>
__device__ double gpu_atomic_add(double* address, const double val) {
#if __CUDA_ARCH__ >= 600
  return atomicAdd(address, val);
#else
  auto address_as_ull = reinterpret_cast<unsigned long long int*>(address);
  unsigned long long int old = *address_as_ull;
  unsigned long long int assumed = 0;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
#endif
}

template<>
__device__ float gpu_atomic_max(float* address, const float val) {
  int* address_as_i = (int*)address;
  int old = *address_as_i;
  int assumed = 0;
  do {
    assumed = old;
    old = atomicCAS(address_as_i, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

template<>
__device__ double gpu_atomic_max(double* address, const double val) {
  unsigned long long int* address_as_i = (unsigned long long int*)address;
  unsigned long long int old = *address_as_i;
  unsigned long long int assumed = 0;
  do {
    assumed = old;
    old = atomicCAS(address_as_i, assumed,
                    __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}

template<typename T, typename U>
__global__ void CastOnGpu(const T* in, U* out, int64_t elem_num) {
  CUDA_1D_KERNEL_LOOP(i, elem_num) { out[i] = static_cast<U>(in[i]); }
}

template<>
__global__ void CastOnGpu<float, half>(const float* in, half* out, int64_t elem_num) {
  CUDA_1D_KERNEL_LOOP(i, elem_num) { out[i] = __float2half(in[i]); }
}

template<>
__global__ void CastOnGpu<half, float>(const half* in, float* out, int64_t elem_num) {
  CUDA_1D_KERNEL_LOOP(i, elem_num) { out[i] = __half2float(in[i]); }
}

template<typename T, typename U>
void CopyElemOnGpu(DeviceCtx* ctx, const T* in_dptr, U* out_dptr, int64_t elem_num) {
  if (std::is_same<T, U>::value) {
    Memcpy<DeviceType::kGPU>(ctx, out_dptr, in_dptr, elem_num * sizeof(T));
  } else {
    CastOnGpu<T, U>
        <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            in_dptr, out_dptr, elem_num);
  }
}

template<>
void CopyElemOnGpu<float, float16>(DeviceCtx* ctx, const float* in_dptr, float16* out_dptr,
                                   int64_t elem_num) {
  CastOnGpu<float, half>
      <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          in_dptr, reinterpret_cast<half*>(out_dptr), elem_num);
}

template<>
void CopyElemOnGpu<float16, float>(DeviceCtx* ctx, const float16* in_dptr, float* out_dptr,
                                   int64_t elem_num) {
  CastOnGpu<half, float>
      <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          reinterpret_cast<const half*>(in_dptr), out_dptr, elem_num);
}

#define INSTANTIATE_COPY_ELEM_ON_GPU(T, U) \
  template void CopyElemOnGpu(DeviceCtx* ctx, const T* in_dptr, U* out_dptr, int64_t elem_num);

#define MAKE_COPY_ELEM_ON_GPU_ENTRY(TPair, UPair) \
  INSTANTIATE_COPY_ELEM_ON_GPU(OF_PP_PAIR_FIRST(TPair), OF_PP_PAIR_FIRST(UPair))

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_COPY_ELEM_ON_GPU_ENTRY, POD_DATA_TYPE_SEQ, POD_DATA_TYPE_SEQ)

}  // namespace oneflow
