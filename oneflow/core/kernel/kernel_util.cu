#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/kernel_util.h"

namespace oneflow {
template<typename FloatingPointType>
class KernelUtil<DeviceType::kGPU, FloatingPointType> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(KernelUtil);
  KernelUtil() = delete;

  static void Memcpy(const KernelCtx& ctx, void* dst, const void* src,
                     size_t sz, hipMemcpyKind kind) {
    CHECK_EQ(hipMemcpyAsync(dst, src, sz, kind, ctx.device_ctx->hip_stream()),
             hipSuccess);
  }

  static void Memset(const KernelCtx& ctx, void* dst, const char value,
                     size_t sz) {
    CHECK_EQ(hipMemsetAsync(dst, value, sz, ctx.device_ctx->hip_stream()),
             hipSuccess);
  }

  static void BlasAxpy(const KernelCtx& ctx, const int n,
                       const FloatingPointType alpha,
                       const FloatingPointType* x, const int incx,
                       FloatingPointType* y, const int incy) {
    cublas_axpy(ctx.device_ctx->cublas_handle(), n, &alpha, x, incx, y, incy);
  }

  static void BlasScal(const KernelCtx& ctx, const int n,
                       const FloatingPointType alpha, FloatingPointType* x,
                       const int incx) {
    cublas_scal(ctx.device_ctx->cublas_handle(), n, &alpha, x, incx);
  }

  static void BlasGemv(const KernelCtx& ctx, const enum CBLAS_TRANSPOSE trans,
                       int m, int n, const FloatingPointType alpha,
                       const FloatingPointType* a, int lda,
                       const FloatingPointType* x, const int incx,
                       const FloatingPointType beta, FloatingPointType* y,
                       const int incy) {
    hipblasOperation_t cublas_trans = CblasTrans2CublasTrans(trans);
    cublas_gemv(ctx.device_ctx->cublas_handle(), cublas_trans, n, m, &alpha, a,
                lda, x, incx, &beta, y, incy);
  }

  static void BlasGemm(const KernelCtx& ctx, const enum CBLAS_ORDER order,
                       const enum CBLAS_TRANSPOSE trans_a,
                       const enum CBLAS_TRANSPOSE trans_b, const int m,
                       const int n, const int k, const FloatingPointType alpha,
                       const FloatingPointType* a, const int lda,
                       const FloatingPointType* b, const int ldb,
                       const FloatingPointType beta, FloatingPointType* c,
                       const int ldc) {
    hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
    hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
    cublas_gemm(ctx.device_ctx->cublas_handle(), cublas_trans_b, cublas_trans_a,
                n, m, k, &alpha, b, ldb, a, lda, &beta, c, ldc);
  }

  static void BlasDot(const KernelCtx& ctx, const int n,
                      const FloatingPointType* x, const int incx,
                      const FloatingPointType* y, const int incy,
                      FloatingPointType* result) {
    cublas_dot(ctx.device_ctx->cublas_handle(), n, x, incx, y, incy, result);
  }

  static void BlasSwap(const KernelCtx& ctx, const int n, FloatingPointType* x,
                       const int incx, FloatingPointType* y, const int incy) {
    cublas_swap(ctx.device_ctx->cublas_handle(), n, x, incx, y, incy);
  }

  static void BlasCopy(const KernelCtx& ctx, const int n,
                       const FloatingPointType* x, const int incx,
                       FloatingPointType* y, const int incy) {
    cublas_copy(ctx.device_ctx->cublas_handle(), n, x, incx, y, incy);
  }

 private:
  static hipblasOperation_t CblasTrans2CublasTrans(CBLAS_TRANSPOSE trans) {
    hipblasOperation_t cublas_trans;
    if (trans == CBLAS_TRANSPOSE::CblasNoTrans) {
      cublas_trans = hipblasOperation_t::HIPBLAS_OP_N;
    } else if (trans == CBLAS_TRANSPOSE::CblasTrans) {
      cublas_trans = hipblasOperation_t::HIPBLAS_OP_T;
    } else if (trans == CBLAS_TRANSPOSE::CblasConjTrans) {
      cublas_trans = hipblasOperation_t::HIPBLAS_OP_C;
    } else {
      // do nothing
    }
    return cublas_trans;
  }
};

INSTANTIATE_GPU_KERNEL_UTIL_CLASS(KernelUtil);

}  // namespace oneflow
