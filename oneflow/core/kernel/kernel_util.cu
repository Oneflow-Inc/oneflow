#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <hipcub/hipcub.hpp>
#include <math.h>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/new_kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void AxpyGpu(const int n, const T alpha, const T* x, const int incx, T* y,
                        const int incy) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i * incy] += alpha * x[i * incx]; }
}

template<typename T>
__global__ void gpu_assign_add(const int64_t n, T* out, const T* in_1) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    if (in_1[i]) { out[i] += in_1[i]; }
  }
}

template<typename T>
__global__ void gpu_assign_add(const int64_t n, T* out, const T* in_1, const T* in_2) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] += in_1[i] + in_2[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i]; }
}
template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i];
  }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6, const T* in_7) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i] + in_7[i];
  }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6, const T* in_7,
                        const T* in_8) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] =
        in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i] + in_7[i] + in_8[i];
  }
}

}  // namespace

#define MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY(func_name, T) hipcub::DeviceReduce::func_name<T*, T*>
DEFINE_STATIC_SWITCH_FUNC(hipError_t, Sum, MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY,
                          MAKE_DATA_TYPE_CTRV_SEQ(FLOATING_DATA_TYPE_SEQ));

#undef MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY

#define KU_IF_METHOD                     \
  template<typename T, typename Derived> \
  void GpuKernelUtilIf<T, Derived>::

KU_IF_METHOD InitializeWithConf(DeviceCtx* ctx, const InitializerConf& initializer_conf,
                                uint32_t random_seed, Blob* blob) {
  WithHostBlobAndStreamSynchronizeEnv(ctx, blob, [&](Blob* host_blob) {
    KernelUtil<DeviceType::kCPU, T>::InitializeWithConf(nullptr, initializer_conf, random_seed,
                                                        host_blob);
  });
}

#define KU_FLOATING_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kGPU, T, typename std::enable_if<IsFloating<T>::value>::type>::

KU_FLOATING_METHOD Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx,
                        T* y, const int incy) {
  cublas_axpy<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx, y, incy);
}
KU_FLOATING_METHOD Axpy(DeviceCtx* ctx, const int n, const T* alpha, const T* x, const int incx,
                        T* y, const int incy) {
  cublas_axpy<T>(ctx->cublas_pmd_handle(), n, alpha, x, incx, y, incy);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0) {
  gpu_add<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, out, in_0);
}
KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1) {
  if (out == in_0) {
    gpu_assign_add<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, out, in_1);
  } else {
    gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, out, in_0, in_1);
  }
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2) {
  if (out == in_0) {
    gpu_assign_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, out, in_1, in_2);
  } else {
    gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, out, in_0, in_1, in_2);
  }
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6, const T* in_7) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6, in_7);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6, const T* in_7, const T* in_8) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6, in_7, in_8);
}

#define KU_INTEGRAL_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kGPU, T, typename std::enable_if<IsIntegral<T>::value>::type>::

KU_INTEGRAL_METHOD Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx,
                        T* y, const int incy) {
  AxpyGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, alpha, x, incx, y, incy);
}

#define INSTANTIATE_KERNEL_UTIL(type_cpp, type_proto)                                \
  template struct GpuKernelUtilIf<type_cpp, KernelUtil<DeviceType::kGPU, type_cpp>>; \
  template struct KernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KERNEL_UTIL, ARITHMETIC_DATA_TYPE_SEQ);

template<typename T, typename U>
__global__ void CastOnGpu(const T* in, U* out, int64_t elem_num) {
  CUDA_1D_KERNEL_LOOP(i, elem_num) { out[i] = static_cast<U>(in[i]); }
}

template<>
__global__ void CastOnGpu<float, half>(const float* in, half* out, int64_t elem_num) {
  const int64_t elem_num_2 = elem_num / 2;
  const auto* in_2 = reinterpret_cast<const float2*>(in);
  auto* out_2 = reinterpret_cast<half2*>(out);
  CUDA_1D_KERNEL_LOOP(i, elem_num_2) { out_2[i] = __float22half2_rn(in_2[i]); }
  if (elem_num % 2 == 1 && blockIdx.x == 0 && threadIdx.x == 0) {
    out[elem_num - 1] = __float2half(in[elem_num - 1]);
  }
}

template<>
__global__ void CastOnGpu<half, float>(const half* in, float* out, int64_t elem_num) {
  const int64_t elem_num_2 = elem_num / 2;
  const auto* in_2 = reinterpret_cast<const half2*>(in);
  auto* out_2 = reinterpret_cast<float2*>(out);
  CUDA_1D_KERNEL_LOOP(i, elem_num_2) { out_2[i] = __half22float2(in_2[i]); }
  if (elem_num % 2 == 1 && blockIdx.x == 0 && threadIdx.x == 0) {
    out[elem_num - 1] = __half2float(in[elem_num - 1]);
  }
}

template<typename T, typename U>
void CopyElemOnGpu(DeviceCtx* ctx, const T* in_dptr, U* out_dptr, int64_t elem_num) {
  if (elem_num == 0) { return; }
  if (std::is_same<T, U>::value) {
    Memcpy<DeviceType::kGPU>(ctx, out_dptr, in_dptr, elem_num * sizeof(T));
  } else {
    CastOnGpu<T, U>
        <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            in_dptr, out_dptr, elem_num);
  }
}

template<>
void CopyElemOnGpu<float, float16>(DeviceCtx* ctx, const float* in_dptr, float16* out_dptr,
                                   int64_t elem_num) {
  if (RoundUp(elem_num, 2) == 0) { return; }
  CastOnGpu<float, half>
      <<<BlocksNum4ThreadsNum(RoundUp(elem_num, 2) / 2), kCudaThreadsNumPerBlock, 0,
         ctx->hip_stream()>>>(in_dptr, reinterpret_cast<half*>(out_dptr), elem_num);
}

template<>
void CopyElemOnGpu<float16, float>(DeviceCtx* ctx, const float16* in_dptr, float* out_dptr,
                                   int64_t elem_num) {
  if (RoundUp(elem_num, 2) == 0) { return; }
  CastOnGpu<half, float>
      <<<BlocksNum4ThreadsNum(RoundUp(elem_num, 2) / 2), kCudaThreadsNumPerBlock, 0,
         ctx->hip_stream()>>>(reinterpret_cast<const half*>(in_dptr), out_dptr, elem_num);
}

#define INSTANTIATE_COPY_ELEM_ON_GPU(T, U) \
  template void CopyElemOnGpu(DeviceCtx* ctx, const T* in_dptr, U* out_dptr, int64_t elem_num);

#define MAKE_COPY_ELEM_ON_GPU_ENTRY(TPair, UPair) \
  INSTANTIATE_COPY_ELEM_ON_GPU(OF_PP_PAIR_FIRST(TPair), OF_PP_PAIR_FIRST(UPair))

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_COPY_ELEM_ON_GPU_ENTRY, POD_DATA_TYPE_SEQ, POD_DATA_TYPE_SEQ)

}  // namespace oneflow
