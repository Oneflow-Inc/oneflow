#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void RsqrtGpu(const int64_t n, T* x, const float epsilon) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = 1.0 / std::sqrt(x[i] + epsilon); }
}

template<typename T>
__global__ void ExpGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::exp(x[i]); }
}

template<typename T>
__global__ void DivGpu(const int64_t n, T* x, const T* alpha_ptr) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = x[i] / (*alpha_ptr); }
}

template<typename T>
__global__ void MulGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y[i]; }
}

hipblasOperation_t CblasTrans2CublasTrans(CBLAS_TRANSPOSE trans) {
  hipblasOperation_t cublas_trans;
  if (trans == CBLAS_TRANSPOSE::CblasNoTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_N;
  } else if (trans == CBLAS_TRANSPOSE::CblasTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_T;
  } else if (trans == CBLAS_TRANSPOSE::CblasConjTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_C;
  } else {
    // do nothing
  }
  return cublas_trans;
}

}  // namespace

template<>
void Memcpy<DeviceType::kGPU>(DeviceCtx* ctx, void* dst, const void* src,
                              size_t sz, hipMemcpyKind kind) {
  CudaCheck(hipMemcpyAsync(dst, src, sz, kind, ctx->hip_stream()));
}

template<>
void Memset<DeviceType::kGPU>(DeviceCtx* ctx, void* dst, const char value,
                              size_t sz) {
  CudaCheck(hipMemsetAsync(dst, value, sz, ctx->hip_stream()));
}

#define MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY(func_name, T) \
  hipcub::DeviceReduce::func_name<T*, T*>
DEFINE_STATIC_SWITCH_FUNC(hipError_t, Sum, MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY,
                          MAKE_DATA_TYPE_CTRV_SEQ(FLOATING_DATA_TYPE_SEQ));

size_t GetTmpSizeForReduceSum(DataType data_type, int64_t sum_elem_num) {
  size_t tmp_storage_size;
  SwitchSum(SwitchCase(data_type), nullptr, tmp_storage_size, nullptr, nullptr,
            sum_elem_num);
  return tmp_storage_size;
}

#undef MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY

#define KU_IF_METHOD                     \
  template<typename T, typename Derived> \
  void GpuKernelUtilIf<T, Derived>::

KU_IF_METHOD Max(DeviceCtx* ctx, const int64_t n, const T* x, T* max_ptr,
                 T* temp_storage, size_t temp_storage_bytes) {
  CudaCheck(hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, x, max_ptr,
                                   n, ctx->hip_stream()));
}
KU_IF_METHOD Sum(DeviceCtx* ctx, const int64_t n, const T* x, T* sum_ptr,
                 T* temp_storage, size_t temp_storage_bytes) {
  CudaCheck(hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes, x, sum_ptr,
                                   n, ctx->hip_stream()));
}

#define KU_FLOATING_METHOD             \
  template<typename T>                 \
  void KernelUtil<DeviceType::kGPU, T, \
                  typename std::enable_if<IsFloating<T>::value>::type>::

KU_FLOATING_METHOD Dot(DeviceCtx* ctx, const int n, const T* x, const int incx,
                       const T* y, const int incy, T* result) {
  cublas_dot<T>(ctx->cublas_pmd_handle(), n, x, incx, y, incy, result);
}
KU_FLOATING_METHOD Copy(DeviceCtx* ctx, const int n, const T* x, const int incx,
                        T* y, const int incy) {
  cublas_copy<T>(ctx->cublas_pmh_handle(), n, x, incx, y, incy);
}
KU_FLOATING_METHOD Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x,
                        const int incx, T* y, const int incy) {
  cublas_axpy<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx, y, incy);
}
KU_FLOATING_METHOD Axpy(DeviceCtx* ctx, const int n, const T* alpha, const T* x,
                        const int incx, T* y, const int incy) {
  cublas_axpy<T>(ctx->cublas_pmd_handle(), n, alpha, x, incx, y, incy);
}
KU_FLOATING_METHOD Scal(DeviceCtx* ctx, const int n, const T alpha, T* x,
                        const int incx) {
  cublas_scal<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx);
}
KU_FLOATING_METHOD Scal(DeviceCtx* ctx, const int n, const T* alpha, T* x,
                        const int incx) {
  cublas_scal<T>(ctx->cublas_pmd_handle(), n, alpha, x, incx);
}
KU_FLOATING_METHOD Gemv(DeviceCtx* ctx, const enum CBLAS_TRANSPOSE trans, int m,
                        int n, const T alpha, const T* a, int lda, const T* x,
                        const int incx, const T beta, T* y, const int incy) {
  hipblasOperation_t cublas_trans = CblasTrans2CublasTrans(trans);
  cublas_gemv<T>(ctx->cublas_pmh_handle(), cublas_trans, n, m, &alpha, a, lda,
                 x, incx, &beta, y, incy);
}
KU_FLOATING_METHOD Gemm(DeviceCtx* ctx, const enum CBLAS_ORDER order,
                        const enum CBLAS_TRANSPOSE trans_a,
                        const enum CBLAS_TRANSPOSE trans_b, const int m,
                        const int n, const int k, const T alpha, const T* a,
                        const int lda, const T* b, const int ldb, const T beta,
                        T* c, const int ldc) {
  hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
  hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
  cublas_gemm<T>(ctx->cublas_pmh_handle(), cublas_trans_b, cublas_trans_a, n, m,
                 k, &alpha, b, ldb, a, lda, &beta, c, ldc);
}

KU_FLOATING_METHOD Exp(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
  ExpGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
              ctx->hip_stream()>>>(n, x, y);
}
KU_FLOATING_METHOD Div(DeviceCtx* ctx, const int64_t n, T* x, const T* alpha) {
  DivGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
              ctx->hip_stream()>>>(n, x, alpha);
}
KU_FLOATING_METHOD Mul(DeviceCtx* ctx, const int64_t n, const T* x, const T* y,
                       T* z) {
  MulGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
              ctx->hip_stream()>>>(n, x, y, z);
}
KU_FLOATING_METHOD Rsqrt(DeviceCtx* ctx, const int64_t n, T* x,
                         const float epsilon) {
  RsqrtGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                ctx->hip_stream()>>>(n, x, epsilon);
}

#define CREATE_FORWARD_TENSOR_AND_ACTIVATION_DESCRIPTOR(mode)               \
  CudnnTensorDesc x_desc(HIPDNN_TENSOR_NCHW, GetDataType<T>::value, n, 1, 1, \
                         1);                                                \
  CudnnTensorDesc y_desc(HIPDNN_TENSOR_NCHW, GetDataType<T>::value, n, 1, 1, \
                         1);                                                \
  CudnnActivationDesc act_desc(mode, HIPDNN_PROPAGATE_NAN, 0.0);

#define FORWARD_COMPUTE_ACTIVATION(mode)                                \
  CREATE_FORWARD_TENSOR_AND_ACTIVATION_DESCRIPTOR(mode);                \
  CudaCheck(hipdnnActivationForward(ctx->cudnn_handle(), act_desc.Get(), \
                                   OnePtr<T>::value, x_desc.Get(), x,   \
                                   ZeroPtr<T>::value, y_desc.Get(), y));

#define CREATE_BACKWARD_TENSOR_AND_ACTIVATION_DESCRIPTOR(mode)               \
  CREATE_FORWARD_TENSOR_AND_ACTIVATION_DESCRIPTOR(mode);                     \
  CudnnTensorDesc dx_desc(HIPDNN_TENSOR_NCHW, GetDataType<T>::value, n, 1, 1, \
                          1);                                                \
  CudnnTensorDesc dy_desc(HIPDNN_TENSOR_NCHW, GetDataType<T>::value, n, 1, 1, 1);

#define BACKWARD_COMPUTE_ACTIVATION(mode)                                \
  CREATE_BACKWARD_TENSOR_AND_ACTIVATION_DESCRIPTOR(mode);                \
  CudaCheck(hipdnnActivationBackward(ctx->cudnn_handle(), act_desc.Get(), \
                                    OnePtr<T>::value, y_desc.Get(), y,   \
                                    dy_desc.Get(), dy, x_desc.Get(), x,  \
                                    ZeroPtr<T>::value, dx_desc.Get(), dx));

KU_FLOATING_METHOD Sigmoid(DeviceCtx* ctx, int64_t n, const T* x, T* y){
    FORWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_SIGMOID)} KU_FLOATING_METHOD
    SigmoidBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y,
                    const T* dy, T* dx) {
  BACKWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_SIGMOID);
}
KU_FLOATING_METHOD TanH(DeviceCtx* ctx, int64_t n, const T* x, T* y) {
  FORWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_TANH);
}
KU_FLOATING_METHOD TanHBackward(DeviceCtx* ctx, const int64_t n, const T* x,
                                const T* y, const T* dy, T* dx) {
  BACKWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_TANH);
}
KU_FLOATING_METHOD Relu(DeviceCtx* ctx, int64_t n, const T* x, T* y) {
  FORWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_RELU);
}
KU_FLOATING_METHOD ReluBackward(DeviceCtx* ctx, const int64_t n, const T* x,
                                const T* y, const T* dy, T* dx) {
  BACKWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_RELU);
}

KU_FLOATING_METHOD InitializeWithConf(DeviceCtx* ctx,
                                      const InitializerConf& initializer_conf,
                                      uint32_t random_seed, Blob* blob) {
  // create temporary host blob store initializer result
  BlobDesc blob_desc = BlobDesc(blob->blob_desc());
  char* host_raw_dptr = nullptr;
  CudaCheck(hipHostMalloc(&host_raw_dptr, blob->TotalByteSize()));
  std::unique_ptr<Blob> host_blob;
  host_blob.reset(
      NewBlob(nullptr, &blob_desc, host_raw_dptr, nullptr, DeviceType::kGPU));
  // synchronous initialize the host blob
  KernelUtil<DeviceType::kCPU, T>::InitializeWithConf(
      nullptr, initializer_conf, random_seed, host_blob.get());
  // asynchronous copy to device
  Memcpy<DeviceType::kGPU>(ctx, blob->mut_dptr(), host_blob->dptr(),
                           blob->ByteSizeOfDataContentField(),
                           hipMemcpyHostToDevice);
  hipStreamSynchronize(ctx->hip_stream());
  CudaCheck(hipHostFree(host_raw_dptr));
}

KU_FLOATING_METHOD InitializeWithDir(DeviceCtx* ctx, int32_t part_id,
                                     int32_t part_num,
                                     const std::string& model_dir, Blob* blob,
                                     const std::string& bn_in_op,
                                     int32_t dim_num, int64_t num_in_each_dim) {
  BlobDesc blob_desc = BlobDesc(blob->blob_desc());
  char* host_raw_dptr = nullptr;
  CudaCheck(hipHostMalloc(&host_raw_dptr, blob->TotalByteSize()));
  std::unique_ptr<Blob> host_blob;
  host_blob.reset(
      NewBlob(nullptr, &blob_desc, host_raw_dptr, nullptr, DeviceType::kGPU));
  KernelUtil<DeviceType::kCPU, T>::InitializeWithDir(
      ctx, part_id, part_num, model_dir, host_blob.get(), bn_in_op, dim_num,
      num_in_each_dim);

  Memcpy<DeviceType::kGPU>(ctx, blob->mut_dptr(), host_blob->dptr(),
                           blob->ByteSizeOfDataContentField(),
                           hipMemcpyHostToDevice);
  hipStreamSynchronize(ctx->hip_stream());
  CudaCheck(hipHostFree(host_raw_dptr));
}

#define INSTANTIATE_KERNEL_UTIL(type_cpp, type_proto)                      \
  template struct GpuKernelUtilIf<type_cpp,                                \
                                  KernelUtil<DeviceType::kGPU, type_cpp>>; \
  template struct KernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KERNEL_UTIL, ARITHMETIC_DATA_TYPE_SEQ);

template<>
__device__ float gpu_atomic_add(float* address, const float val) {
  return atomicAdd(address, val);
}

template<>
__device__ double gpu_atomic_add(double* address, const double val) {
  auto address_as_ull = reinterpret_cast<unsigned long long int*>(address);
  unsigned long long int old = *address_as_ull;
  unsigned long long int assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

}  // namespace oneflow
