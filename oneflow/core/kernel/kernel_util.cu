/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <math.h>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/new_kernel_util.h"

namespace oneflow {

#define KU_IF_METHOD                     \
  template<typename T, typename Derived> \
  void GpuKernelUtilIf<T, Derived>::

KU_IF_METHOD InitializeWithConf(ep::Stream* stream, const InitializerConf& initializer_conf,
                                uint32_t random_seed, Blob* blob) {
  WithHostBlobAndStreamSynchronizeEnv(stream, blob, [&](Blob* host_blob) {
    KernelUtil<DeviceType::kCPU, T>::InitializeWithConf(nullptr, initializer_conf, random_seed,
                                                        host_blob);
  });
}

#define KU_FLOATING_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kCUDA, T, typename std::enable_if<IsFloating<T>::value>::type>::

#define KU_INTEGRAL_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kCUDA, T, typename std::enable_if<IsIntegral<T>::value>::type>::

#define INSTANTIATE_KERNEL_UTIL(type_cpp, type_proto)                                 \
  template struct GpuKernelUtilIf<type_cpp, KernelUtil<DeviceType::kCUDA, type_cpp>>; \
  template struct KernelUtil<DeviceType::kCUDA, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KERNEL_UTIL, ARITHMETIC_DATA_TYPE_SEQ);

}  // namespace oneflow
