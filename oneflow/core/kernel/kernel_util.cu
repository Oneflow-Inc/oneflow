#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <hipcub/hipcub.hpp>
#include <math.h>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/new_kernel_util.h"

namespace oneflow {

#define MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY(func_name, T) hipcub::DeviceReduce::func_name<T*, T*>
DEFINE_STATIC_SWITCH_FUNC(hipError_t, Sum, MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY,
                          MAKE_DATA_TYPE_CTRV_SEQ(FLOATING_DATA_TYPE_SEQ));

#undef MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY

#define KU_IF_METHOD                     \
  template<typename T, typename Derived> \
  void GpuKernelUtilIf<T, Derived>::

KU_IF_METHOD InitializeWithConf(DeviceCtx* ctx, const InitializerConf& initializer_conf,
                                uint32_t random_seed, Blob* blob) {
  WithHostBlobAndStreamSynchronizeEnv(ctx, blob, [&](Blob* host_blob) {
    KernelUtil<DeviceType::kCPU, T>::InitializeWithConf(nullptr, initializer_conf, random_seed,
                                                        host_blob);
  });
}

#define KU_FLOATING_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kGPU, T, typename std::enable_if<IsFloating<T>::value>::type>::

#define KU_INTEGRAL_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kGPU, T, typename std::enable_if<IsIntegral<T>::value>::type>::

#define INSTANTIATE_KERNEL_UTIL(type_cpp, type_proto)                                \
  template struct GpuKernelUtilIf<type_cpp, KernelUtil<DeviceType::kGPU, type_cpp>>; \
  template struct KernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KERNEL_UTIL, ARITHMETIC_DATA_TYPE_SEQ);

template<typename T, typename U>
__global__ void CastOnGpu(const T* in, U* out, int64_t elem_num) {
  CUDA_1D_KERNEL_LOOP(i, elem_num) { out[i] = static_cast<U>(in[i]); }
}

template<>
__global__ void CastOnGpu<float, half>(const float* in, half* out, int64_t elem_num) {
  const int64_t elem_num_2 = elem_num / 2;
  const auto* in_2 = reinterpret_cast<const float2*>(in);
  auto* out_2 = reinterpret_cast<half2*>(out);
  CUDA_1D_KERNEL_LOOP(i, elem_num_2) { out_2[i] = __float22half2_rn(in_2[i]); }
  if (elem_num % 2 == 1 && blockIdx.x == 0 && threadIdx.x == 0) {
    out[elem_num - 1] = __float2half(in[elem_num - 1]);
  }
}

template<>
__global__ void CastOnGpu<half, float>(const half* in, float* out, int64_t elem_num) {
  const int64_t elem_num_2 = elem_num / 2;
  const auto* in_2 = reinterpret_cast<const half2*>(in);
  auto* out_2 = reinterpret_cast<float2*>(out);
  CUDA_1D_KERNEL_LOOP(i, elem_num_2) { out_2[i] = __half22float2(in_2[i]); }
  if (elem_num % 2 == 1 && blockIdx.x == 0 && threadIdx.x == 0) {
    out[elem_num - 1] = __half2float(in[elem_num - 1]);
  }
}

template<typename T, typename U>
void CopyElemOnGpu(DeviceCtx* ctx, const T* in_dptr, U* out_dptr, int64_t elem_num) {
  if (elem_num == 0) { return; }
  if (std::is_same<T, U>::value) {
    Memcpy<DeviceType::kGPU>(ctx, out_dptr, in_dptr, elem_num * sizeof(T));
  } else {
    CastOnGpu<T, U>
        <<<BlocksNum4ThreadsNum(elem_num), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            in_dptr, out_dptr, elem_num);
  }
}

template<>
void CopyElemOnGpu<float, float16>(DeviceCtx* ctx, const float* in_dptr, float16* out_dptr,
                                   int64_t elem_num) {
  if (RoundUp(elem_num, 2) == 0) { return; }
  CastOnGpu<float, half>
      <<<BlocksNum4ThreadsNum(RoundUp(elem_num, 2) / 2), kCudaThreadsNumPerBlock, 0,
         ctx->hip_stream()>>>(in_dptr, reinterpret_cast<half*>(out_dptr), elem_num);
}

template<>
void CopyElemOnGpu<float16, float>(DeviceCtx* ctx, const float16* in_dptr, float* out_dptr,
                                   int64_t elem_num) {
  if (RoundUp(elem_num, 2) == 0) { return; }
  CastOnGpu<half, float>
      <<<BlocksNum4ThreadsNum(RoundUp(elem_num, 2) / 2), kCudaThreadsNumPerBlock, 0,
         ctx->hip_stream()>>>(reinterpret_cast<const half*>(in_dptr), out_dptr, elem_num);
}

#define INSTANTIATE_COPY_ELEM_ON_GPU(T, U) \
  template void CopyElemOnGpu(DeviceCtx* ctx, const T* in_dptr, U* out_dptr, int64_t elem_num);

#define MAKE_COPY_ELEM_ON_GPU_ENTRY(TPair, UPair) \
  INSTANTIATE_COPY_ELEM_ON_GPU(OF_PP_PAIR_FIRST(TPair), OF_PP_PAIR_FIRST(UPair))

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_COPY_ELEM_ON_GPU_ENTRY, POD_DATA_TYPE_SEQ, POD_DATA_TYPE_SEQ)

}  // namespace oneflow
