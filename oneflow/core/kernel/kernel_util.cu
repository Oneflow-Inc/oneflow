#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void RsqrtGpu(const int64_t n, T* x, const float epsilon) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = 1.0 / std::sqrt(x[i] + epsilon); }
}

template<typename T>
__global__ void ExpGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::exp(x[i]); }
}

template<typename T>
__global__ void DivGpu(const int64_t n, T* x, const T* alpha_ptr) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = x[i] / (*alpha_ptr); }
}

template<typename T>
__global__ void MulGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y[i]; }
}

hipblasOperation_t CblasTrans2CublasTrans(CBLAS_TRANSPOSE trans) {
  hipblasOperation_t cublas_trans;
  if (trans == CBLAS_TRANSPOSE::CblasNoTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_N;
  } else if (trans == CBLAS_TRANSPOSE::CblasTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_T;
  } else if (trans == CBLAS_TRANSPOSE::CblasConjTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_C;
  } else {
    // do nothing
  }
  return cublas_trans;
}

}  // namespace

template<>
void Memcpy<DeviceType::kGPU>(DeviceCtx* ctx, void* dst, const void* src,
                              size_t sz, hipMemcpyKind kind) {
  CudaCheck(hipMemcpyAsync(dst, src, sz, kind, ctx->hip_stream()));
}

template<>
void Memset<DeviceType::kGPU>(DeviceCtx* ctx, void* dst, const char value,
                              size_t sz) {
  CudaCheck(hipMemsetAsync(dst, value, sz, ctx->hip_stream()));
}

template<typename T>
struct KernelUtil<DeviceType::kGPU, T> final {
  static void Dot(DeviceCtx* ctx, const int n, const T* x, const int incx,
                  const T* y, const int incy, T* result) {
    cublas_dot<T>(ctx->cublas_pmd_handle(), n, x, incx, y, incy, result);
  }
  static void Copy(DeviceCtx* ctx, const int n, const T* x, const int incx,
                   T* y, const int incy) {
    cublas_copy<T>(ctx->cublas_pmh_handle(), n, x, incx, y, incy);
  }
  static void Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x,
                   const int incx, T* y, const int incy) {
    cublas_axpy<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx, y, incy);
  }
  static void Axpy(DeviceCtx* ctx, const int n, const T* alpha, const T* x,
                   const int incx, T* y, const int incy) {
    cublas_axpy<T>(ctx->cublas_pmd_handle(), n, alpha, x, incx, y, incy);
  }
  static void Scal(DeviceCtx* ctx, const int n, const T alpha, T* x,
                   const int incx) {
    cublas_scal<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx);
  }
  static void Scal(DeviceCtx* ctx, const int n, const T* alpha, T* x,
                   const int incx) {
    cublas_scal<T>(ctx->cublas_pmd_handle(), n, alpha, x, incx);
  }
  static void Rsqrt(DeviceCtx* ctx, const int64_t n, T* x,
                    const float epsilon) {
    RsqrtGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                  ctx->hip_stream()>>>(n, x, epsilon);
  static void Gemv(DeviceCtx* ctx, const enum CBLAS_TRANSPOSE trans, int m,
                   int n, const T alpha, const T* a, int lda, const T* x,
                   const int incx, const T beta, T* y, const int incy) {
    hipblasOperation_t cublas_trans = CblasTrans2CublasTrans(trans);
    cublas_gemv<T>(ctx->cublas_pmh_handle(), cublas_trans, n, m, &alpha, a, lda,
                   x, incx, &beta, y, incy);
  }
  static void Gemm(DeviceCtx* ctx, const enum CBLAS_ORDER order,
                   const enum CBLAS_TRANSPOSE trans_a,
                   const enum CBLAS_TRANSPOSE trans_b, const int m, const int n,
                   const int k, const T alpha, const T* a, const int lda,
                   const T* b, const int ldb, const T beta, T* c,
                   const int ldc) {
    hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
    hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
    cublas_gemm<T>(ctx->cublas_pmh_handle(), cublas_trans_b, cublas_trans_a, n,
                   m, k, &alpha, b, ldb, a, lda, &beta, c, ldc);
  }

  static void Max(DeviceCtx* ctx, const int64_t n, const T* x, T* max_ptr,
                  T* temp_storage, size_t temp_storage_bytes) {
    hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, x, max_ptr, n,
                           ctx->hip_stream());
  }
  static void Exp(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
    ExpGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                ctx->hip_stream()>>>(n, x, y);
  }
  static void Sum(DeviceCtx* ctx, const int64_t n, const T* x, T* sum_ptr,
                  T* temp_storage, size_t temp_storage_bytes) {
    hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes, x, sum_ptr, n,
                           ctx->hip_stream());
  }
  static void Div(DeviceCtx* ctx, const int64_t n, T* x, const T* alpha) {
    DivGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                ctx->hip_stream()>>>(n, x, alpha);
  }
  static void Mul(DeviceCtx* ctx, const int64_t n, const T* x, const T* y,
                  T* z) {
    MulGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
                ctx->hip_stream()>>>(n, x, y, z);
  }

#define CREATE_FORWARD_TENSOR_AND_ACTIVATION_DESCRIPTOR(mode)               \
  CudnnTensorDesc x_desc(HIPDNN_TENSOR_NCHW, GetDataType<T>::value, n, 1, 1, \
                         1);                                                \
  CudnnTensorDesc y_desc(HIPDNN_TENSOR_NCHW, GetDataType<T>::value, n, 1, 1, \
                         1);                                                \
  CudnnActivationDesc act_desc(mode, HIPDNN_PROPAGATE_NAN, 0.0);

#define FORWARD_COMPUTE_ACTIVATION(mode)                                \
  CREATE_FORWARD_TENSOR_AND_ACTIVATION_DESCRIPTOR(mode);                \
  CudaCheck(hipdnnActivationForward(ctx->cudnn_handle(), act_desc.Get(), \
                                   OnePtr<T>::value, x_desc.Get(), x,   \
                                   ZeroPtr<T>::value, y_desc.Get(), y));

#define CREATE_BACKWARD_TENSOR_AND_ACTIVATION_DESCRIPTOR(mode)               \
  CREATE_FORWARD_TENSOR_AND_ACTIVATION_DESCRIPTOR(mode);                     \
  CudnnTensorDesc dx_desc(HIPDNN_TENSOR_NCHW, GetDataType<T>::value, n, 1, 1, \
                          1);                                                \
  CudnnTensorDesc dy_desc(HIPDNN_TENSOR_NCHW, GetDataType<T>::value, n, 1, 1, 1);

#define BACKWARD_COMPUTE_ACTIVATION(mode)                                \
  CREATE_BACKWARD_TENSOR_AND_ACTIVATION_DESCRIPTOR(mode);                \
  CudaCheck(hipdnnActivationBackward(ctx->cudnn_handle(), act_desc.Get(), \
                                    OnePtr<T>::value, y_desc.Get(), y,   \
                                    dy_desc.Get(), dy, x_desc.Get(), x,  \
                                    ZeroPtr<T>::value, dx_desc.Get(), dx));

  static void Sigmoid(DeviceCtx* ctx, int64_t n, const T* x, T* y) {
    FORWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_SIGMOID)
  }
  static void SigmoidBackward(DeviceCtx* ctx, const int64_t n, const T* x,
                              const T* y, const T* dy, T* dx) {
    BACKWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_SIGMOID);
  }
  static void TanH(DeviceCtx* ctx, int64_t n, const T* x, T* y) {
    FORWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_TANH);
  }
  static void TanHBackward(DeviceCtx* ctx, const int64_t n, const T* x,
                           const T* y, const T* dy, T* dx) {
    BACKWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_TANH);
  }
  static void Relu(DeviceCtx* ctx, int64_t n, const T* x, T* y) {
    FORWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_RELU);
  }
  static void ReluBackward(DeviceCtx* ctx, const int64_t n, const T* x,
                           const T* y, const T* dy, T* dx) {
    BACKWARD_COMPUTE_ACTIVATION(HIPDNN_ACTIVATION_RELU);
  }

  static void Initialize(DeviceCtx* ctx,
                         const InitializerConf& initializer_conf,
                         uint32_t random_seed, Blob* blob) {
    // create temporary host blob store initializer result
    BlobDesc blob_desc = BlobDesc(blob->blob_desc());
    char* host_raw_dptr = nullptr;
    CudaCheck(hipHostMalloc(&host_raw_dptr, blob->TotalByteSize()));
    std::unique_ptr<Blob> host_blob;
    host_blob.reset(
        NewBlob(nullptr, &blob_desc, host_raw_dptr, nullptr, DeviceType::kGPU));
    // synchronous initialize the host blob
    KernelUtil<DeviceType::kCPU, T>::Initialize(nullptr, initializer_conf,
                                                random_seed, host_blob.get());
    // asynchronous copy to device
    Memcpy<DeviceType::kGPU>(ctx, blob->mut_dptr(), host_blob->dptr(),
                             blob->ByteSizeOfDataContentField(),
                             hipMemcpyHostToDevice);
    hipStreamSynchronize(ctx->hip_stream());
    CudaCheck(hipHostFree(host_raw_dptr));
  }

  static void Initialize(DeviceCtx* ctx, int32_t part_id, int32_t part_num,
                         const std::string& model_dir, Blob* blob,
                         const std::string& bn_in_op, int32_t dim_num,
                         int64_t num_in_each_dim) {
    BlobDesc blob_desc = BlobDesc(blob->blob_desc());
    char* host_raw_dptr = nullptr;
    CudaCheck(hipHostMalloc(&host_raw_dptr, blob->TotalByteSize()));
    std::unique_ptr<Blob> host_blob;
    host_blob.reset(
        NewBlob(nullptr, &blob_desc, host_raw_dptr, nullptr, DeviceType::kGPU));
    KernelUtil<DeviceType::kCPU, T>::Initialize(
        ctx, part_id, part_num, model_dir, host_blob.get(), bn_in_op, dim_num,
        num_in_each_dim);

    Memcpy<DeviceType::kGPU>(ctx, blob->mut_dptr(), host_blob->dptr(),
                             blob->ByteSizeOfDataContentField(),
                             hipMemcpyHostToDevice);
    hipStreamSynchronize(ctx->hip_stream());
    CudaCheck(hipHostFree(host_raw_dptr));
  }
};

template<>
__device__ float gpu_atomic_add(float* address, const float val) {
  return atomicAdd(address, val);
}

template<>
__device__ double gpu_atomic_add(double* address, const double val) {
  auto address_as_ull = reinterpret_cast<unsigned long long int*>(address);
  unsigned long long int old = *address_as_ull;
  unsigned long long int assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

#define INSTANTIATE_KERNEL_UTIL(type_cpp, type_proto) \
  template struct KernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ);

#define DEFINE_INT_KERNEL_UTIL(T, type_proto)                                 \
  template void KernelUtil<DeviceType::kGPU, T>::Sum(                         \
      DeviceCtx* ctx, const int64_t n, const T* x, T* sum_ptr,                \
      T* temp_storage, size_t temp_storage_bytes);                            \
  template void KernelUtil<DeviceType::kGPU, T>::Max(                         \
      DeviceCtx* ctx, const int64_t n, const T* x, T* max_ptr,                \
      T* temp_storage, size_t temp_storage_bytes);                            \
  template void KernelUtil<DeviceType::kGPU, T>::Relu(                        \
      DeviceCtx* ctx, const int64_t n, const T* x, T* y);                     \
  template void KernelUtil<DeviceType::kGPU, T>::ReluBackward(                \
      DeviceCtx* ctx, const int64_t n, const T* x, const T* y, const T* dy,   \
      T* dx);                                                                 \
  template<>                                                                  \
  void KernelUtil<DeviceType::kGPU, T>::Axpy(                                 \
      DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx, \
      T* y, const int incy) {                                                 \
    FOR_RANGE(int, i, 0, n) {                                                 \
      *y += alpha * *x;                                                       \
      x += incx;                                                              \
      y += incy;                                                              \
    }                                                                         \
  }

OF_PP_FOR_EACH_TUPLE(DEFINE_INT_KERNEL_UTIL, INT_DATA_TYPE_SEQ);

}  // namespace oneflow
