#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <math.h>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void RsqrtGpu(const int64_t n, T* x, const float epsilon) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = 1.0 / std::sqrt(x[i] + epsilon); }
}

template<typename T>
__global__ void ExpGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::exp(x[i]); }
}

template<typename T>
__global__ void DivByConstParaPtrGpu(const int64_t n, T* x, const T* alpha_ptr) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = x[i] / (*alpha_ptr); }
}

template<typename T>
__global__ void DivGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] / y[i]; }
}

template<typename T>
__global__ void DivByConstParaGpu(const int64_t n, T* x, const T alpha) {
  CUDA_1D_KERNEL_LOOP(i, n) { x[i] = x[i] / alpha; }
}

template<typename T>
__global__ void ReplicateGpu(const int64_t n, T* y, const T* x) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = *x; }
}

template<typename T>
__global__ void MulGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y[i]; }
}

template<typename T>
__global__ void MulByScalarGpu(const int64_t n, const T* x, const T* y, T* z) {
  CUDA_1D_KERNEL_LOOP(i, n) { z[i] = x[i] * y[0]; }
}

template<typename T>
__global__ void ReciprocalGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = static_cast<T>(1.0) / x[i]; }
}

template<typename T>
__global__ void SquareGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = x[i] * x[i]; }
}

template<typename T>
__global__ void SqrtGpu(const int64_t n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::sqrt(x[i]); }
}

template<typename T>
__global__ void AxpyGpu(const int n, const T alpha, const T* x, const int incx, T* y,
                        const int incy) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i * incy] += alpha * x[i * incx]; }
}

template<typename T>
__global__ void SigmoidForwardGpu(const int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = 1.0 / (1.0 + std::exp(-x[i])); }
}

template<typename T>
__global__ void SigmoidBackwardGpu(const int n, const T* y, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = dy[i] * y[i] * (1.0 - y[i]); }
}

template<typename T>
__global__ void TanHForwardGpu(const int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::tanh(x[i]); }
}

template<typename T>
__global__ void TanHBackwardGpu(const int n, const T* y, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = dy[i] * (1.0 - y[i] * y[i]); }
}

template<typename T>
__global__ void ReluForwardGpu(const int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = x[i] > 0 ? x[i] : 0; }
}

template<typename T>
__global__ void ReluBackwardGpu(const int n, const T* y, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = y[i] > 0 ? dy[i] : 0; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i]; }
}
template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i]; }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i];
  }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6, const T* in_7) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] = in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i] + in_7[i];
  }
}

template<typename T>
__global__ void gpu_add(const int64_t n, T* out, const T* in_0, const T* in_1, const T* in_2,
                        const T* in_3, const T* in_4, const T* in_5, const T* in_6, const T* in_7,
                        const T* in_8) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] =
        in_0[i] + in_1[i] + in_2[i] + in_3[i] + in_4[i] + in_5[i] + in_6[i] + in_7[i] + in_8[i];
  }
}

template<typename T>
__global__ void gpu_set(const T value, T* addr) {
  *addr = value;
}

hipblasOperation_t CblasTrans2CublasTrans(CBLAS_TRANSPOSE trans) {
  hipblasOperation_t cublas_trans;
  if (trans == CBLAS_TRANSPOSE::CblasNoTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_N;
  } else if (trans == CBLAS_TRANSPOSE::CblasTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_T;
  } else if (trans == CBLAS_TRANSPOSE::CblasConjTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_C;
  } else {
    // do nothing
  }
  return cublas_trans;
}

const int32_t kMaxDim = OF_PP_SEQ_SIZE(DIM_SEQ);

struct Int32Array {
  int32_t val[kMaxDim];
};

struct Int64Array {
  int64_t val[kMaxDim];
};

__device__ void ComputeOffset(const int32_t num_axis, const int64_t* x_dims,
                              const int32_t* permutation, int64_t* x_strides) {
  int64_t buff[kMaxDim];
  int64_t cur_stride = 1;
  for (int32_t i = num_axis - 1; i >= 0; --i) {
    buff[i] = cur_stride;
#if __CUDA_ARCH__ >= 350
    cur_stride *= __ldg(x_dims + i);
#else
    cur_stride *= x_dims[i];
#endif
  }
  for (int32_t i = 0; i < num_axis; ++i) {
#if __CUDA_ARCH__ >= 350
    x_strides[i] = buff[__ldg(permutation + i)];
#else
    x_strides[i] = buff[permutation[i]];
#endif
  }
}

template<typename T>
__global__ void CopyColsRegionGpu(const int64_t row_num, const int64_t col_num, const T* x,
                                  const int64_t x_col_offset, const int64_t x_lda, T* y,
                                  const int64_t y_col_offset, const int64_t y_lda) {
  CUDA_1D_KERNEL_LOOP(index, row_num * col_num) {
    const int64_t i = index / col_num;
    const int64_t j = index % col_num;
    y[i * y_lda + y_col_offset + j] = x[i * x_lda + x_col_offset + j];
  }
}

__device__ int64_t GetXIndex(const int32_t num_axis, const int64_t* y_shape,
                             const int64_t* x_strides, int64_t y_idx) {
  int64_t x_idx = 0;
  for (int32_t i = num_axis - 1; i >= 0 && y_idx > 0; --i) {
    x_idx += (y_idx % y_shape[i]) * x_strides[i];
    y_idx /= y_shape[i];
  }
  return x_idx;
}

template<typename T>
__global__ void TransposeGpu(const int32_t num_axis, const Int64Array x_shape,
                             const Int64Array y_shape, const Int32Array permutation,
                             const int64_t elem_cnt, const T* x, T* y) {
  __shared__ int64_t x_strides[kMaxDim];
  __shared__ int64_t x_dims_shared[kMaxDim];
  __shared__ int64_t y_dims_shared[kMaxDim];
  __shared__ int32_t perm_shared[kMaxDim];
  const int32_t tid = threadIdx.x;
  if (tid < num_axis) {
    x_dims_shared[tid] = x_shape.val[tid];
    y_dims_shared[tid] = y_shape.val[tid];
    perm_shared[tid] = permutation.val[tid];
  }
  __syncthreads();
  if (tid == 0) { ComputeOffset(num_axis, x_dims_shared, perm_shared, x_strides); }
  __syncthreads();
  CUDA_1D_KERNEL_LOOP(y_idx, elem_cnt) {
    const int64_t x_idx = GetXIndex(num_axis, y_dims_shared, x_strides, y_idx);
#if __CUDA_ARCH__ >= 350
    y[y_idx] = __ldg(x + x_idx);
#else
    y[y_idx] = x[x_idx];
#endif
  }
}

template<typename T, T (*reduce_core_func)(const T, const T)>
__device__ void MatrixShrinkCols(const size_t row_num, const size_t thread_col_num, const T* x,
                                 const size_t x_col_num, const size_t x_lda, T* y,
                                 const size_t y_col_num, const size_t y_lda) {
  const size_t thread_num = blockDim.x * gridDim.x;
  const size_t total_shrink_scale = thread_col_num / y_col_num;
  CUDA_1D_KERNEL_LOOP(index, row_num * thread_col_num) {
    const int32_t thread_col = index % thread_col_num;
    if (((index / thread_num) % total_shrink_scale) != thread_col / y_col_num) { continue; }
    const int32_t row = index / thread_col_num;
    const int32_t col = thread_col % y_col_num;
    const int32_t x_start = row * x_lda + col;
    const int32_t x_end = row * x_lda + x_col_num;
    T reduced = x[x_start];
    for (int32_t x_index = x_start + y_col_num; x_index < x_end; x_index += y_col_num) {
      reduced = reduce_core_func(reduced, x[x_index]);
    }
    y[row * y_lda + col] = reduced;
  }
}

template<typename T, T (*reduce_core_func)(const T, const T), size_t shift_size = 2>
__global__ void MatrixRowReduceGpu(const size_t row_num, const size_t col_num, const T* x, T* y,
                                   T* temp_storage, size_t temp_col_num) {
  const size_t temp_lda = temp_col_num;
  MatrixShrinkCols<T, reduce_core_func>(row_num, temp_lda, x, col_num, col_num, temp_storage,
                                        temp_col_num, temp_lda);
  __syncthreads();
  while (temp_col_num > (1 << shift_size)) {
    size_t new_temp_col_num = temp_col_num >> shift_size;
    MatrixShrinkCols<T, reduce_core_func>(row_num, temp_lda, temp_storage, temp_col_num, temp_lda,
                                          temp_storage, new_temp_col_num, temp_lda);
    temp_col_num = new_temp_col_num;
    __syncthreads();
  }
  MatrixShrinkCols<T, reduce_core_func>(row_num, temp_lda, temp_storage, temp_col_num, temp_lda, y,
                                        1, 1);
}

template<typename T, T (*reduce_core_func)(const T, const T), size_t shift_size = 2>
void MatrixRowReduce(DeviceCtx* ctx, const size_t row_num, const size_t col_num, const T* x, T* y,
                     void* temp_storage, const size_t temp_storage_bytes) {
  CHECK_NOTNULL(temp_storage);
  CHECK_GT(temp_storage_bytes / sizeof(T), row_num);
  const size_t temp_col_num_shift =
      std::floor(std::log2(std::min(temp_storage_bytes / sizeof(T) / row_num, col_num)));
  const size_t temp_col_num = std::min(static_cast<size_t>(kCudaThreadsNumPerBlock),
                                       static_cast<size_t>(1 << temp_col_num_shift));
  MatrixRowReduceGpu<T, reduce_core_func>
      <<<BlocksNum4ThreadsNum(row_num * temp_col_num), kCudaThreadsNumPerBlock, 0,
         ctx->hip_stream()>>>(row_num, col_num, x, y, static_cast<T*>(temp_storage), temp_col_num);
}

}  // namespace

template<>
void Memcpy<DeviceType::kGPU>(DeviceCtx* ctx, void* dst, const void* src, size_t sz,
                              hipMemcpyKind kind) {
  CudaCheck(hipMemcpyAsync(dst, src, sz, kind, ctx->hip_stream()));
}

template<>
void Memset<DeviceType::kGPU>(DeviceCtx* ctx, void* dst, const char value, size_t sz) {
  CudaCheck(hipMemsetAsync(dst, value, sz, ctx->hip_stream()));
}

#define MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY(func_name, T) hipcub::DeviceReduce::func_name<T*, T*>
DEFINE_STATIC_SWITCH_FUNC(hipError_t, Sum, MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY,
                          MAKE_DATA_TYPE_CTRV_SEQ(FLOATING_DATA_TYPE_SEQ));

size_t GetTmpSizeForReduceSum(DataType data_type, int64_t sum_elem_num) {
  size_t tmp_storage_size;
  SwitchSum(SwitchCase(data_type), nullptr, tmp_storage_size, nullptr, nullptr, sum_elem_num);
  return tmp_storage_size;
}

#undef MAKE_CUB_DEVICE_REDUCE_SWITCH_ENTRY

// create temporary host blob store initializer result
#define BEFORE_CPU_INITIALIZE()                                     \
  RtBlobDesc blob_desc(blob->blob_desc().blob_desc_proto());        \
  char* host_raw_dptr = nullptr;                                    \
  CudaCheck(hipHostMalloc(&host_raw_dptr, blob->TotalByteSize())); \
  std::unique_ptr<Blob> host_blob;                                  \
  host_blob.reset(new Blob(nullptr, &blob_desc, host_raw_dptr));

// asynchronous copy to device
#define AFTER_CPU_INITIALIZE()                                                          \
  Memcpy<DeviceType::kGPU>(ctx, blob->mut_dptr(), host_blob->dptr(),                    \
                           blob->ByteSizeOfDataContentField(), hipMemcpyHostToDevice); \
  CudaCheck(hipStreamSynchronize(ctx->hip_stream()));                                 \
  CudaCheck(hipHostFree(host_raw_dptr));

#define KU_IF_METHOD                     \
  template<typename T, typename Derived> \
  void GpuKernelUtilIf<T, Derived>::

KU_IF_METHOD Max(DeviceCtx* ctx, const int64_t n, const T* x, T* max_ptr, T* temp_storage,
                 size_t temp_storage_bytes) {
  CudaCheck(
      hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, x, max_ptr, n, ctx->hip_stream()));
}
KU_IF_METHOD Sum(DeviceCtx* ctx, const int64_t n, const T* x, T* sum_ptr, T* temp_storage,
                 size_t temp_storage_bytes) {
  CudaCheck(
      hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes, x, sum_ptr, n, ctx->hip_stream()));
}
KU_IF_METHOD CopyColsRegion(DeviceCtx* ctx, const int64_t row_num, const int64_t col_num,
                            const T* x, const int64_t x_col_offset, const int64_t x_lda, T* y,
                            const int64_t y_col_offset, const int64_t y_lda) {
  CopyColsRegionGpu<T>
      <<<BlocksNum4ThreadsNum(row_num * col_num), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          row_num, col_num, x, x_col_offset, x_lda, y, y_col_offset, y_lda);
}
KU_IF_METHOD RowMax(DeviceCtx* ctx, const int64_t row_num, const int64_t col_num, const T* x, T* y,
                    void* temp_storage, const size_t temp_storage_bytes) {
  MatrixRowReduce<T, ReduceCoreMax>(ctx, row_num, col_num, x, y, temp_storage, temp_storage_bytes);
}
KU_IF_METHOD RowSum(DeviceCtx* ctx, const int64_t row_num, const int64_t col_num, const T* x, T* y,
                    void* temp_storage, const size_t temp_storage_bytes) {
  MatrixRowReduce<T, ReduceCoreAdd>(ctx, row_num, col_num, x, y, temp_storage, temp_storage_bytes);
}
KU_IF_METHOD Transpose(DeviceCtx* ctx, const int32_t num_axis, const Shape& x_shape,
                       const Shape& y_shape, const PbRf<int32_t>& permutation,
                       const int64_t elem_cnt, const T* x, T* y) {
  CHECK_LE(num_axis, kMaxDim);
  Int64Array x_shape_struct;
  Int64Array y_shape_struct;
  Int32Array perm_struct;
  FOR_RANGE(int32_t, i, 0, num_axis) {
    x_shape_struct.val[i] = x_shape.At(i);
    y_shape_struct.val[i] = y_shape.At(i);
    perm_struct.val[i] = permutation[i];
  }
  TransposeGpu<T>
      <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          num_axis, x_shape_struct, y_shape_struct, perm_struct, elem_cnt, x, y);
}

KU_IF_METHOD InitializeWithConf(DeviceCtx* ctx, const InitializerConf& initializer_conf,
                                uint32_t random_seed, Blob* blob) {
  BEFORE_CPU_INITIALIZE();
  // synchronous initialize the host blob
  KernelUtil<DeviceType::kCPU, T>::InitializeWithConf(nullptr, initializer_conf, random_seed,
                                                      host_blob.get());
  AFTER_CPU_INITIALIZE();
}
KU_IF_METHOD InitializeWithConf(DeviceCtx* ctx, const InitializerConf& initializer_conf,
                                uint32_t random_seed, Blob* blob, const std::string& data_format) {
  BEFORE_CPU_INITIALIZE();
  // synchronous initialize the host blob
  KernelUtil<DeviceType::kCPU, T>::InitializeWithConf(nullptr, initializer_conf, random_seed,
                                                      host_blob.get(), data_format);
  AFTER_CPU_INITIALIZE();
}
KU_IF_METHOD InitializeWithDir(DeviceCtx* ctx, int32_t part_id, int32_t part_num,
                               const std::string& model_dir, Blob* blob,
                               const std::string& bn_in_op, int32_t dim_num,
                               int64_t num_in_each_dim) {
  BEFORE_CPU_INITIALIZE();
  KernelUtil<DeviceType::kCPU, T>::InitializeWithDir(
      ctx, part_id, part_num, model_dir, host_blob.get(), bn_in_op, dim_num, num_in_each_dim);
  AFTER_CPU_INITIALIZE();
}
KU_IF_METHOD Set(DeviceCtx* ctx, const T value, T* addr) {
  gpu_set<T><<<1, 1, 0, ctx->hip_stream()>>>(value, addr);
}
KU_IF_METHOD Replicate(DeviceCtx* ctx, const int64_t n, T* y, const T* x) {
  ReplicateGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, y, x);
}

#define KU_FLOATING_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kGPU, T, typename std::enable_if<IsFloating<T>::value>::type>::

KU_FLOATING_METHOD Dot(DeviceCtx* ctx, const int n, const T* x, const int incx, const T* y,
                       const int incy, T* result) {
  cublas_dot<T>(ctx->cublas_pmd_handle(), n, x, incx, y, incy, result);
}
KU_FLOATING_METHOD Copy(DeviceCtx* ctx, const int n, const T* x, const int incx, T* y,
                        const int incy) {
  cublas_copy<T>(ctx->cublas_pmh_handle(), n, x, incx, y, incy);
}
KU_FLOATING_METHOD Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx,
                        T* y, const int incy) {
  cublas_axpy<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx, y, incy);
}
KU_FLOATING_METHOD Axpy(DeviceCtx* ctx, const int n, const T* alpha, const T* x, const int incx,
                        T* y, const int incy) {
  cublas_axpy<T>(ctx->cublas_pmd_handle(), n, alpha, x, incx, y, incy);
}
KU_FLOATING_METHOD Scal(DeviceCtx* ctx, const int n, const T alpha, T* x, const int incx) {
  cublas_scal<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx);
}
KU_FLOATING_METHOD Scal(DeviceCtx* ctx, const int n, const T* alpha, T* x, const int incx) {
  cublas_scal<T>(ctx->cublas_pmd_handle(), n, alpha, x, incx);
}
KU_FLOATING_METHOD Gemv(DeviceCtx* ctx, const enum CBLAS_TRANSPOSE trans, int m, int n,
                        const T alpha, const T* a, int lda, const T* x, const int incx,
                        const T beta, T* y, const int incy) {
  hipblasOperation_t cublas_trans = CblasTrans2CublasTrans(trans);
  cublas_gemv<T>(ctx->cublas_pmh_handle(), cublas_trans, n, m, &alpha, a, lda, x, incx, &beta, y,
                 incy);
}
KU_FLOATING_METHOD Gemm(DeviceCtx* ctx, const enum CBLAS_ORDER order,
                        const enum CBLAS_TRANSPOSE trans_a, const enum CBLAS_TRANSPOSE trans_b,
                        const int m, const int n, const int k, const T alpha, const T* a,
                        const int lda, const T* b, const int ldb, const T beta, T* c,
                        const int ldc) {
  hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
  hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
  cublas_gemm<T>(ctx->cublas_pmh_handle(), cublas_trans_b, cublas_trans_a, n, m, k, &alpha, b, ldb,
                 a, lda, &beta, c, ldc);
}

KU_FLOATING_METHOD Exp(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
  ExpGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}
KU_FLOATING_METHOD Div(DeviceCtx* ctx, const int64_t n, T* x, const T* alpha) {
  DivByConstParaPtrGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, alpha);
}
KU_FLOATING_METHOD Div(DeviceCtx* ctx, const int64_t n, T* x, const T alpha) {
  DivByConstParaGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, alpha);
}
KU_FLOATING_METHOD Div(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, T* z) {
  DivGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z);
}
KU_FLOATING_METHOD Mul(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, T* z) {
  MulGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z);
}
KU_FLOATING_METHOD MulByScalar(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, T* z) {
  MulByScalarGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y, z);
}
KU_FLOATING_METHOD Reciprocal(DeviceCtx* ctx, const int n, const T* x, T* y) {
  ReciprocalGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}
KU_FLOATING_METHOD Square(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
  SquareGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}
KU_FLOATING_METHOD Sqrt(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
  SqrtGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}
KU_FLOATING_METHOD Rsqrt(DeviceCtx* ctx, const int64_t n, T* x, const float epsilon) {
  RsqrtGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, epsilon);
}

KU_FLOATING_METHOD Sigmoid(DeviceCtx* ctx, int64_t n, const T* x, T* y) {
  SigmoidForwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}

KU_FLOATING_METHOD SigmoidBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y,
                                   const T* dy, T* dx) {
  SigmoidBackwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, y, dy, dx);
}

KU_FLOATING_METHOD TanH(DeviceCtx* ctx, int64_t n, const T* x, T* y) {
  TanHForwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}

KU_FLOATING_METHOD TanHBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y,
                                const T* dy, T* dx) {
  TanHBackwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, y, dy, dx);
}

KU_FLOATING_METHOD Relu(DeviceCtx* ctx, int64_t n, const T* x, T* y) {
  ReluForwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
}

KU_FLOATING_METHOD ReluBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y,
                                const T* dy, T* dx) {
  ReluBackwardGpu<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, y, dy, dx);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0) {
  gpu_add<T>
      <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, out, in_0);
}
KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6, const T* in_7) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6, in_7);
}

KU_FLOATING_METHOD Addition(DeviceCtx* ctx, const int64_t n, T* out, const T* in_0, const T* in_1,
                            const T* in_2, const T* in_3, const T* in_4, const T* in_5,
                            const T* in_6, const T* in_7, const T* in_8) {
  gpu_add<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, out, in_0, in_1, in_2, in_3, in_4, in_5, in_6, in_7, in_8);
}

#define KU_INTEGRAL_METHOD \
  template<typename T>     \
  void KernelUtil<DeviceType::kGPU, T, typename std::enable_if<IsIntegral<T>::value>::type>::

KU_INTEGRAL_METHOD Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx,
                        T* y, const int incy) {
  AxpyGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, alpha, x, incx, y, incy);
}

#define INSTANTIATE_KERNEL_UTIL(type_cpp, type_proto)                                \
  template struct GpuKernelUtilIf<type_cpp, KernelUtil<DeviceType::kGPU, type_cpp>>; \
  template struct KernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KERNEL_UTIL, ARITHMETIC_DATA_TYPE_SEQ);

template<>
__device__ float gpu_atomic_add(float* address, const float val) {
  return atomicAdd(address, val);
}

template<>
__device__ double gpu_atomic_add(double* address, const double val) {
  auto address_as_ull = reinterpret_cast<unsigned long long int*>(address);
  unsigned long long int old = *address_as_ull;
  unsigned long long int assumed = 0;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

template<>
__device__ float gpu_atomic_max(float* address, const float val) {
  int* address_as_i = (int*)address;
  int old = *address_as_i;
  int assumed = 0;
  do {
    assumed = old;
    old = atomicCAS(address_as_i, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

template<>
__device__ double gpu_atomic_max(double* address, const double val) {
  unsigned long long int* address_as_i = (unsigned long long int*)address;
  unsigned long long int old = *address_as_i;
  unsigned long long int assumed = 0;
  do {
    assumed = old;
    old = atomicCAS(address_as_i, assumed,
                    __double_as_longlong(fmaxf(val, __longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}

}  // namespace oneflow
