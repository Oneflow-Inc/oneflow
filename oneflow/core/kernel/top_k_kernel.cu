#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/top_k_kernel.h"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/common/util.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include "oneflow/core/kernel/top_k_heap_selection.cuh"
#include "oneflow/core/kernel/gpu_bitonic_sort.cuh"

#include <iostream>

namespace oneflow {

#define MAX_POWER 16

int32_t PowOf2Floor(int32_t val) {
  int32_t ret = 0;
  for (int32_t i = 0; i <= MAX_POWER; i++) {
    ret = std::pow(2, i);
    if (val < ret) {
      return ret / 2;
    } else if (val == ret) {
      return ret;
    }
  }
  return -1;
}

int32_t PowOf2Ceil(int32_t val) {
  int32_t ret = 0;
  for (int32_t i = 0; i <= MAX_POWER; i++) {
    ret = std::pow(2, i);
    if (val <= ret) { return ret; }
  }
  return -1;
}

template<typename T>
__global__ void HeapTopKKernel(const T* in, const int32_t instance_num, const int32_t instance_size,
                               const int32_t k, const int32_t heap_size, const int32_t init_index,
                               const T init_value, T* out) {
  extern __shared__ char smem[];
  auto* shared_entries = reinterpret_cast<Entry<T>*>(smem);

  const T* input = in + blockIdx.x * instance_size;
  auto heap = Heap<T>(shared_entries + threadIdx.x * heap_size, heap_size, init_index, init_value);
  // Divide elements to be sorted into disjoint sets (# of sets == # of heaps).
  // Each thread in the thread block manipulate one heap to select top heap_size entries from
  // corresponding set
  for (int32_t i = threadIdx.x; i < instance_size; i += blockDim.x) {
    auto entry = Entry<T>(i, input[i]);
    if (entry > heap[0]) { heap.ReplaceRoot(entry); }
  }
  // Merge all heaps to a unified, sorted array
  bitonicSort<Entry<T>, EntryGTComp<Entry<T>>>(shared_entries, blockDim.x * heap_size);
  T* output = out + blockIdx.x * (blockDim.x * heap_size);
  for (int32_t i = 0; i < blockDim.x * heap_size; ++i) {
    output[i] = static_cast<T>(shared_entries[i].GetIndex());
  }

  // Write top_k elements in sorted array to output
  // int32_t* output = out + blockIdx.x * k;
  // for (int32_t i = 0; i < k; ++i) { output[i] = shared_entries[i].GetIndex(); }
}

template<typename T>
struct TopKKernelUtil<DeviceType::kGPU, T> {
  static void Forward(DeviceCtx* ctx, const T* in, const int32_t instance_num,
                      const int32_t instance_size, const int32_t k, const bool sorted,
                      int32_t* fw_buf, T* out) {
    CHECK(fw_buf == nullptr);
    // if (instance_size <= 1000 || k == instance_size || k > 512) {
    if (false) {
      TODO();
    } else {
      // Use as many heaps as possible (# of heaps == # of threads in thread block).
      // Limitation 1, max shared memory: 48KB
      // We also need heap_size * num_heap to be pow-of-2 which is necessary for bitonic sort
      // implemented in our system
      const int32_t heap_size = PowOf2Ceil(k);
      std::cout << "heap_size: " << heap_size << std::endl;
      const int32_t heap_byte_size = heap_size * sizeof(Entry<T>);
      std::cout << "heap_byte_size: " << heap_byte_size << std::endl;
      int32_t num_heap = PowOf2Floor(kCudaMaxSharedMemoryByteSize / heap_byte_size);
      CHECK_GT(num_heap, 0);
      // Limitation 2: # of threads in a thread block
      if (num_heap > kCudaThreadsNumPerBlock) { num_heap = kCudaThreadsNumPerBlock; }
      std::cout << "num_heap: " << num_heap << std::endl;

      // Calculate shared memory size in thread block
      const int64_t smem_size = num_heap * heap_byte_size;
      CHECK_LE(smem_size, kCudaMaxSharedMemoryByteSize);
      std::cout << "smem_size: " << smem_size << std::endl;

      std::cout << "max int32_t: " << GetMaxVal<int32_t>() << std::endl;
      std::cout << "min T: " << GetMinVal<T>() << std::endl;

      HeapTopKKernel<T><<<instance_num, num_heap, smem_size, ctx->hip_stream()>>>(
          in, instance_num, instance_size, k, heap_size, GetMaxVal<int32_t>(), GetMinVal<T>(), out);
    }
  }
};

#define INSTANTIATE_TOP_K_KERNEL_UTIL(type_cpp, type_proto) \
  template struct TopKKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_TOP_K_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)
#undef INSTANTIATE_TOP_K_KERNEL_UTIL

}  // namespace oneflow
