#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/softmax_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void SoftmaxForwardMaxGpu(const int64_t n, const int64_t w,
                                     const T* out, T* tmp) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    T max_value = out[i * w];
    for (int64_t j = 0; j < w; ++j) {
      max_value = max_value > out[i * w + j] ? max_value : out[i * w + j];
    }
    tmp[i] = max_value;
  }
}

template<typename T>
__global__ void SoftmaxForwardSumGpu(const int64_t n, const int64_t w,
                                     const T* out, T* tmp) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    T sum_value = 0;
    for (int64_t j = 0; j < w; ++j) { sum_value += out[i * w + j]; }
    tmp[i] = sum_value;
  }
}

template<typename T>
__global__ void SoftmaxSubGpu(const int64_t n, const int64_t w, T* matrix,
                              const T* vector) {
  CUDA_1D_KERNEL_LOOP(i, n * w) { matrix[i] -= vector[i / w]; }
}

template<typename T>
__global__ void SoftmaxBackwardDotGpu(const int64_t n, const int64_t w,
                                      const T* out, const T* out_diff, T* tmp) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    T dot_result = 0;
    for (int64_t j = 0; j < w; ++j) {
      dot_result += out[i * w + j] * out_diff[i * w + j];
    }
    tmp[i] = dot_result;
  }
}

}  // namespace

template<typename T>
class SoftmaxKernelUtil<DeviceType::kGPU, T> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(SoftmaxKernelUtil);
  SoftmaxKernelUtil() = delete;

  static void ForwardMax(DeviceCtx* ctx, const int64_t n, const int64_t w,
                         const T* out, T* tmp) {
    SoftmaxForwardMaxGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock,
                              0, ctx->hip_stream()>>>(n, w, out, tmp);
  }

  static void ForwardSum(DeviceCtx* ctx, const int64_t n, const int64_t w,
                         const T* out, T* tmp) {
    SoftmaxForwardSumGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock,
                              0, ctx->hip_stream()>>>(n, w, out, tmp);
  }

  static void Sub(DeviceCtx* ctx, const int64_t n, const int64_t w, T* matrix,
                  const T* vector) {
    SoftmaxSubGpu<T><<<BlocksNum4ThreadsNum(n * w), kCudaThreadsNumPerBlock, 0,
                       ctx->hip_stream()>>>(n, w, matrix, vector);
  }

  static void BackwardDot(DeviceCtx* ctx, const int64_t n, const int64_t w,
                          const T* out, const T* out_diff, T* tmp) {
    SoftmaxBackwardDotGpu<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx->hip_stream()>>>(n, w, out, out_diff, tmp);
  }
};

#ifdef USE_CUDNN
template<typename T>
CudnnSoftmaxKernel<T>::CudnnSoftmaxKernel() {
  CudaCheck(hipdnnCreateTensorDescriptor(&this->in_desc_));
  CudaCheck(hipdnnCreateTensorDescriptor(&this->out_desc_));
}

template<typename T>
CudnnSoftmaxKernel<T>::~CudnnSoftmaxKernel() {
  CudaCheck(hipdnnDestroyTensorDescriptor(this->in_desc_));
  CudaCheck(hipdnnDestroyTensorDescriptor(this->out_desc_));
}

template<typename T>
void CudnnSoftmaxKernel<T>::ForwardDataContent(
    const KernelCtx& ctx,
    std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  const Blob* in_blob = BnInOp2Blob("in");
  Blob* out_blob = BnInOp2Blob("out");

  CudaCheck(hipdnnSetTensor4dDescriptor(
      this->in_desc_, HIPDNN_TENSOR_NCHW, CudnnDataType<T>::type,
      in_blob->shape().At(0), in_blob->shape().At(1), in_blob->shape().At(2),
      in_blob->shape().At(3)));
  CudaCheck(hipdnnSetTensor4dDescriptor(
      this->out_desc_, HIPDNN_TENSOR_NCHW, CudnnDataType<T>::type,
      out_blob->shape().At(0), out_blob->shape().At(1), out_blob->shape().At(2),
      out_blob->shape().At(3)));

  CudaCheck(hipdnnSoftmaxForward(
      ctx.device_ctx->cudnn_handle(), HIPDNN_SOFTMAX_ACCURATE,
      HIPDNN_SOFTMAX_MODE_CHANNEL, CudnnDataType<T>::one, this->in_desc_,
      in_blob->dptr<T>(), CudnnDataType<T>::zero, this->out_desc_,
      out_blob->mut_dptr<T>()));
}

template<typename T>
void CudnnSoftmaxKernel<T>::BackwardDataContent(
    const KernelCtx& ctx,
    std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  const Blob* out_blob = BnInOp2Blob("out");
  const Blob* out_diff_blob = BnInOp2Blob("out_diff");
  Blob* in_diff_blob = BnInOp2Blob("in_diff");

  Memset<DeviceType::kGPU>(ctx.device_ctx, in_diff_blob->mut_dptr(), 0,
                           in_diff_blob->ByteSizeOfDataContentField());

  CudaCheck(hipdnnSoftmaxBackward(
      ctx.device_ctx->cudnn_handle(), HIPDNN_SOFTMAX_ACCURATE,
      HIPDNN_SOFTMAX_MODE_CHANNEL, CudnnDataType<T>::one, this->out_desc_,
      out_blob->dptr<T>(), this->out_desc_, out_diff_blob->dptr<T>(),
      CudnnDataType<T>::zero, this->in_desc_, in_diff_blob->mut_dptr<T>()));
}

#define INSTANTIATE_SOFTMAX_KERNEL(type_cpp, type_proto) \
  template class CudnnSoftmaxKernel<type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_SOFTMAX_KERNEL, FLOATING_DATA_TYPE_SEQ)
#endif  // USE_CUDNN

#define INSTANTIATE_SOFTMAX_KERNEL_UTIL(type_cpp, type_proto) \
  template class SoftmaxKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_SOFTMAX_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
