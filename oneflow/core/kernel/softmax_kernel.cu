#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/softmax_kernel.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void SoftmaxSubGpu(const int64_t n, const int64_t w, T* matrix, const T* vector) {
  CUDA_1D_KERNEL_LOOP(i, n * w) { matrix[i] -= vector[i / w]; }
}

template<typename T>
__global__ void SoftmaxDivGpu(const int64_t n, const int64_t w, T* matrix, const T* vector) {
  CUDA_1D_KERNEL_LOOP(i, n * w) { matrix[i] /= vector[i / w]; }
}

}  // namespace

template<typename T>
struct SoftmaxKernelUtil<DeviceType::kGPU, T> {
  static void Sub(DeviceCtx* ctx, const int64_t n, const int64_t w, T* matrix, const T* vector) {
    SoftmaxSubGpu<T>
        <<<BlocksNum4ThreadsNum(n * w), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            n, w, matrix, vector);
  }

  static void Div(DeviceCtx* ctx, const int64_t n, const int64_t w, T* matrix, const T* vector) {
    SoftmaxDivGpu<T>
        <<<BlocksNum4ThreadsNum(n * w), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            n, w, matrix, vector);
  }
};

#define INSTANTIATE_SOFTMAX_KERNEL_UTIL(type_cpp, type_proto) \
  template struct SoftmaxKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_SOFTMAX_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
