#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/resize_nearest_neighbor_kernel.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void ResizeNearestNeighborForward(const int64_t nthreads, const T* in_dptr,
                                             const int64_t channel_num, const int64_t height,
                                             const int64_t width, const int64_t new_height,
                                             const int64_t new_width, const float scale_h,
                                             const float scale_w, const bool align_corners,
                                             T* out_dptr) {
  const int64_t new_area = new_height * new_width;
  const int64_t channel_area = channel_num * height * width;
  const int64_t channel_new_area = channel_num * new_height * new_width;
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t h = (index / new_width) % new_height;
    const int64_t w = index % new_width;
    const int64_t c = (index / new_area) % channel_num;
    const int64_t n = index / channel_new_area;
    const int64_t in_h = min((align_corners) ? static_cast<int64_t>(roundf(h * scale_h))
                                             : static_cast<int64_t>(floorf(h * scale_h)),
                             height - 1);
    const int64_t in_w = min((align_corners) ? static_cast<int64_t>(roundf(w * scale_w))
                                             : static_cast<int64_t>(floorf(w * scale_w)),
                             width - 1);
    out_dptr[index] = in_dptr[n * channel_area + (c * height + in_h) * width + in_w];
  }
}

template<typename T>
__global__ void ResizeNearestNeighborBackward(const int64_t nthreads, const T* out_diff_dptr,
                                              const int64_t channel_num, const int64_t height,
                                              const int64_t width, const int64_t new_height,
                                              const int64_t new_width, const float scale_h,
                                              const float scale_w, const bool align_corners,
                                              T* in_diff_dptr) {
  const int64_t area = height * width;
  const int64_t new_area = new_height * new_width;
  const int64_t channel_area = channel_num * height * width;
  const int64_t channel_new_area = channel_num * new_height * new_width;
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t h = (index / new_width) % new_height;
    const int64_t w = index % new_width;
    const int64_t c = (index / new_area) % channel_num;
    const int64_t n = index / channel_new_area;
    const int64_t in_h = min((align_corners) ? static_cast<int64_t>(roundf(h * scale_h))
                                             : static_cast<int64_t>(floorf(h * scale_h)),
                             height - 1);
    const int64_t in_w = min((align_corners) ? static_cast<int64_t>(roundf(w * scale_w))
                                             : static_cast<int64_t>(floorf(w * scale_w)),
                             width - 1);
    gpu_atomic_add(in_diff_dptr + n * channel_area + (c * height + in_h) * width + in_w,
                   out_diff_dptr[index]);
  }
}

}  // namespace

template<typename T>
struct ResizeNearestNeighborUtil<DeviceType::kGPU, T> {
  static void Forward(const KernelCtx& ctx, const ResizeNearestNeighborKernelConf& kernel_conf,
                      const bool align_corners, const Blob* in_blob, Blob* out_blob) {
    const int64_t elem_cnt = out_blob->shape().elem_cnt();

    ResizeNearestNeighborForward<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                      ctx.device_ctx->hip_stream()>>>(
        elem_cnt, in_blob->dptr<T>(), in_blob->shape().At(1), in_blob->shape().At(2),
        in_blob->shape().At(3), out_blob->shape().At(2), out_blob->shape().At(3),
        kernel_conf.scale_h(), kernel_conf.scale_w(), align_corners, out_blob->mut_dptr<T>());
  }

  static void Backward(const KernelCtx& ctx, const ResizeNearestNeighborKernelConf& kernel_conf,
                       const bool align_corners, const Blob* out_diff_blob, Blob* in_diff_blob) {
    const int64_t elem_cnt = out_diff_blob->shape().elem_cnt();
    ResizeNearestNeighborBackward<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                       ctx.device_ctx->hip_stream()>>>(
        elem_cnt, out_diff_blob->dptr<T>(), in_diff_blob->shape().At(1),
        in_diff_blob->shape().At(2), in_diff_blob->shape().At(3), out_diff_blob->shape().At(2),
        out_diff_blob->shape().At(3), kernel_conf.scale_h(), kernel_conf.scale_w(), align_corners,
        in_diff_blob->mut_dptr<T>());
  }
};

#define INSTANTIATE_RESIZE_NEAREST_NEIGHBOR_KERNEL_UTIL(type_cpp, type_proto) \
  template class ResizeNearestNeighborUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_RESIZE_NEAREST_NEIGHBOR_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ);

}  // namespace oneflow
