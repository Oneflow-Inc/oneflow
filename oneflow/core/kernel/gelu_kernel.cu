#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/gelu_kernel.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void GeluForwardGpu(const int64_t n, const T* x, const T inv_sqrt2, T* y) {
  UNIMPLEMENTED();
}

template<typename T>
__global__ void GeluBackwardGpu(const int64_t n, const T* x, const T* dy, const T inv_sqrt2,
                                const T coef, T* dx) {
  UNIMPLEMENTED();
}

template<>
__global__ void GeluForwardGpu(const int64_t n, const float* x, const float inv_sqrt2, float* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = 0.5f * x[i] * (1.0f + erff(inv_sqrt2 * x[i])); }
}

template<>
__global__ void GeluBackwardGpu(const int64_t n, const float* x, const float* dy,
                                const float inv_sqrt2, const float coef, float* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    dx[i] =
        0.5f * (1.0f + erff(inv_sqrt2 * x[i]) + x[i] * coef * expf(-0.5f * x[i] * x[i])) * dy[i];
  }
}

template<>
__global__ void GeluForwardGpu(const int64_t n, const double* x, const double inv_sqrt2,
                               double* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = 0.5 * x[i] * (1.0 + erf(inv_sqrt2 * x[i])); }
}

template<>
__global__ void GeluBackwardGpu(const int64_t n, const double* x, const double* dy,
                                const double inv_sqrt2, const double coef, double* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    dx[i] = 0.5 * (1.0 + erf(inv_sqrt2 * x[i]) + x[i] * coef * exp(-0.5 * x[i] * x[i])) * dy[i];
  }
}

}  // namespace

template<typename T>
struct GeluKernelUtil<DeviceType::kGPU, T> {
  static void GeluForward(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
    const T inv_sqrt2 = sqrt(0.5);
    GeluForwardGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, x, inv_sqrt2, y);
  }

  static void GeluBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* dy, T* dx) {
    const T inv_sqrt2 = sqrt(0.5);
    const T coef = sqrt(2.0 / acos(-1.0));
    GeluBackwardGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, x, dy, inv_sqrt2, coef, dx);
  }
};

#define INSTANTIATE_GELU_KERNEL_UTIL(type_cpp, type_proto) \
  template struct GeluKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GELU_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
