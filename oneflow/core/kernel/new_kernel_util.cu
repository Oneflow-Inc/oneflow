#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <math.h>
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/new_kernel_util.h"

namespace oneflow {

#define HALF_CHECK_LOG                      \
  printf("use half need nvcc arch >= 530"); \
  assert(false);

/*
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
#else
  HALF_CHECK_LOG
#endif // __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
*/

namespace {

__inline__ __device__ half hone() { return __float2half(1.0); }

__inline__ __device__ half hzero() { return __float2half(0.0); }

__inline__ half float16_2half(float16 x) {
  // TODO: Potential loss of accuracy
  half* ret = reinterpret_cast<half*>(&x);
  return *ret;
}

__inline__ float16 half2float16(half x) {
  // TODO: Potential loss of accuracy
  float16* ret = reinterpret_cast<float16*>(&x);
  return *ret;
}

template<typename T>
__global__ void gpu_set(const T value, T* addr) {
  *addr = value;
}

template<typename T>
__global__ void SigmoidForwardGpu(const int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = 1.0 / (1.0 + std::exp(-x[i])); }
}

template<typename T>
__global__ void SigmoidBackwardGpu(const int n, const T* y, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = dy[i] * y[i] * (1.0 - y[i]); }
}

template<typename T>
__global__ void TanHForwardGpu(const int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = std::tanh(x[i]); }
}

template<typename T>
__global__ void TanHBackwardGpu(const int n, const T* y, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = dy[i] * (1.0 - y[i] * y[i]); }
}

template<typename T>
__global__ void ReluForwardGpu(const int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = x[i] > 0 ? x[i] : 0; }
}

template<typename T>
__global__ void ReluBackwardGpu(const int n, const T* y, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = y[i] > 0 ? dy[i] : 0; }
}

__global__ void SigmoidForwardGpu(const int n, const half* x, half* y) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = __hdiv(hone(), __hadd(hone(), hexp(__hneg(x[i])))); }
#else
  HALF_CHECK_LOG
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__) */
}

__global__ void SigmoidBackwardGpu(const int n, const half* y, const half* dy, half* dx) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = __hmul(dy[i], __hmul(y[i], __hsub(hone(), y[i]))); }
#else
  HALF_CHECK_LOG
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__) */
}

__global__ void TanHForwardGpu(const int n, const half* x, half* y) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) {
    half ex = hexp(x[i]);
    half e_x = hexp(__hneg(x[i]));
    y[i] = __hdiv(__hsub(ex, e_x), __hadd(ex, e_x));
  }
#else
  HALF_CHECK_LOG
#endif  // __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
}

__global__ void TanHBackwardGpu(const int n, const half* y, const half* dy, half* dx) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = __hmul(dy[i], __hsub(hone(), __hmul(y[i], y[i]))); }
#else
  HALF_CHECK_LOG
#endif  // __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
}

__global__ void ReluForwardGpu(const int n, const half* x, half* y) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) {
    if (__hgt(x[i], hzero())) {
      y[i] = x[i];
    } else {
      y[i] = hzero();
    }
  }
#else
  HALF_CHECK_LOG
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__) */
}

__global__ void ReluBackwardGpu(const int n, const half* y, const half* dy, half* dx) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  half zero = __float2half(0.0);
  CUDA_1D_KERNEL_LOOP(i, n) {
    if (__hgt(y[i], zero)) {
      dx[i] = dy[i];
    } else {
      dx[i] = zero;
    }
  }
#else
  HALF_CHECK_LOG
#endif  // __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
}

__global__ void AxpyHalfGpu(const int n, const half alpha, const half* x, const int incx, half* y,
                            const int incy) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) { y[i * incy] = __hfma(alpha, x[i * incx], y[i * incy]); }
#else
  HALF_CHECK_LOG
#endif  // __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
}

__global__ void ScalHalfGpu(const int n, const half alpha, half* x, const int incx) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, n) { x[i * incx] = __hmul(alpha, x[i * incx]); }
#else
  HALF_CHECK_LOG
#endif  // __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
}

__global__ void Float2HalfGpu(const int n, const float* src, half* dst) {
  CUDA_1D_KERNEL_LOOP(i, n) { dst[i] = __float2half(src[i]); }
}

__global__ void Half2FloatGpu(const int n, const half* src, float* dst) {
  CUDA_1D_KERNEL_LOOP(i, n) { dst[i] = __half2float(src[i]); }
}

hipblasOperation_t CblasTrans2CublasTrans(CBLAS_TRANSPOSE trans) {
  hipblasOperation_t cublas_trans;
  if (trans == CBLAS_TRANSPOSE::CblasNoTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_N;
  } else if (trans == CBLAS_TRANSPOSE::CblasTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_T;
  } else if (trans == CBLAS_TRANSPOSE::CblasConjTrans) {
    cublas_trans = hipblasOperation_t::HIPBLAS_OP_C;
  } else {
    // do nothing
  }
  return cublas_trans;
}

template<typename T>
void InitializeWithConfGpu(DeviceCtx* ctx, const InitializerConf& initializer_conf,
                           uint32_t random_seed, Blob* blob, const std::string& data_format) {
  BEFORE_CPU_INITIALIZE();
  // synchronous initialize the host blob
  NewKernelUtil<DeviceType::kCPU, T>::InitializeWithConf(nullptr, initializer_conf, random_seed,
                                                         host_blob.get(), data_format);
  AFTER_CPU_INITIALIZE();
}

template<typename T>
void InitializeWithDirGpu(DeviceCtx* ctx, int32_t part_id, int32_t part_num,
                          const std::string& model_dir, Blob* blob, const std::string& bn_in_op,
                          int32_t dim_num, int64_t num_in_each_dim) {
  BEFORE_CPU_INITIALIZE();
  NewKernelUtil<DeviceType::kCPU, T>::InitializeWithDir(
      ctx, part_id, part_num, model_dir, host_blob.get(), bn_in_op, dim_num, num_in_each_dim);
  AFTER_CPU_INITIALIZE();
}

}  // namespace

// GPU && Floating
template<typename T>
struct NewKernelUtilIf<DeviceType::kGPU, T, typename std::enable_if<IsFloating<T>::value>::type> {
  static void OFGemm(DeviceCtx* ctx, enum CBLAS_TRANSPOSE trans_a, enum CBLAS_TRANSPOSE trans_b,
                     const int m, const int n, const int k, const T alpha, const T* a, const T* b,
                     const T beta, T* c) {
    const int lda = (trans_a == CblasNoTrans) ? k : m;
    const int ldb = (trans_b == CblasNoTrans) ? n : k;
    const int ldc = n;

    FloatingNewKernelUtilIf<DeviceType::kGPU, T>::Gemm(ctx, CblasRowMajor, trans_a, trans_b, m, n,
                                                       k, alpha, a, lda, b, ldb, beta, c, ldc);
  }
  static void InitializeWithConf(DeviceCtx* ctx, const InitializerConf& initializer_conf,
                                 uint32_t random_seed, Blob* blob, const std::string& data_format) {
    InitializeWithConfGpu<T>(ctx, initializer_conf, random_seed, blob, data_format);
  }
  static void InitializeWithDir(DeviceCtx* ctx, int32_t part_id, int32_t part_num,
                                const std::string& model_dir, Blob* blob,
                                const std::string& bn_in_op, int32_t dim_num,
                                int64_t num_in_each_dim) {
    InitializeWithDirGpu<T>(ctx, part_id, part_num, model_dir, blob, bn_in_op, dim_num,
                            num_in_each_dim);
  }
  static void Sigmoid(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
    SigmoidForwardGpu<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
  }
  static void SigmoidBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, const T* dy,
                              T* dx) {
    SigmoidBackwardGpu<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, y, dy, dx);
  }
  static void TanH(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
    TanHForwardGpu<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
  }
  static void TanHBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, const T* dy,
                           T* dx) {
    TanHBackwardGpu<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, y, dy, dx);
  }
  static void Relu(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
    ReluForwardGpu<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, y);
  }
  static void ReluBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, const T* dy,
                           T* dx) {
    ReluBackwardGpu<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, y, dy, dx);
  }
  static void Set(DeviceCtx* ctx, const T value, T* addr) {
    gpu_set<T><<<1, 1, 0, ctx->hip_stream()>>>(value, addr);
  }
  static void Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx, T* y,
                   const int incy) {
    cublas_axpy<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx, y, incy);
  }
  static void Scal(DeviceCtx* ctx, const int n, const T alpha, T* x, const int incx) {
    cublas_scal<T>(ctx->cublas_pmh_handle(), n, &alpha, x, incx);
  }
};

// GPU && Integral
template<typename T>
struct NewKernelUtilIf<DeviceType::kGPU, T, typename std::enable_if<IsIntegral<T>::value>::type> {
  static void InitializeWithConf(DeviceCtx* ctx, const InitializerConf& initializer_conf,
                                 uint32_t random_seed, Blob* blob, const std::string& data_format) {
    InitializeWithConfGpu<T>(ctx, initializer_conf, random_seed, blob, data_format);
  }
  static void InitializeWithDir(DeviceCtx* ctx, int32_t part_id, int32_t part_num,
                                const std::string& model_dir, Blob* blob,
                                const std::string& bn_in_op, int32_t dim_num,
                                int64_t num_in_each_dim) {
    InitializeWithDirGpu<T>(ctx, part_id, part_num, model_dir, blob, bn_in_op, dim_num,
                            num_in_each_dim);
  }
  static void Set(DeviceCtx* ctx, const T value, T* addr) {
    gpu_set<T><<<1, 1, 0, ctx->hip_stream()>>>(value, addr);
  }
};

// GPU && Float16
template<typename T>
struct NewKernelUtilIf<DeviceType::kGPU, T, typename std::enable_if<IsFloat16<T>::value>::type> {
  static void OFGemm(DeviceCtx* ctx, enum CBLAS_TRANSPOSE trans_a, enum CBLAS_TRANSPOSE trans_b,
                     const int m, const int n, const int k, const T alpha, const T* a, const T* b,
                     const T beta, T* c) {
    const int lda = (trans_a == CblasNoTrans) ? k : m;
    const int ldb = (trans_b == CblasNoTrans) ? n : k;
    const int ldc = n;

    Float16NewKernelUtilIf<DeviceType::kGPU, T>::HGemm(ctx, CblasRowMajor, trans_a, trans_b, m, n,
                                                       k, alpha, a, lda, b, ldb, beta, c, ldc);
  }
  static void InitializeWithConf(DeviceCtx* ctx, const InitializerConf& initializer_conf,
                                 uint32_t random_seed, Blob* blob, const std::string& data_format) {
    InitializeWithConfGpu<T>(ctx, initializer_conf, random_seed, blob, data_format);
  }
  static void InitializeWithDir(DeviceCtx* ctx, int32_t part_id, int32_t part_num,
                                const std::string& model_dir, Blob* blob,
                                const std::string& bn_in_op, int32_t dim_num,
                                int64_t num_in_each_dim) {
    InitializeWithDirGpu<T>(ctx, part_id, part_num, model_dir, blob, bn_in_op, dim_num,
                            num_in_each_dim);
  }
  static void Sigmoid(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
    SigmoidForwardGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, reinterpret_cast<const half*>(x), reinterpret_cast<half*>(y));
  }
  static void SigmoidBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, const T* dy,
                              T* dx) {
    SigmoidBackwardGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, reinterpret_cast<const half*>(y), reinterpret_cast<const half*>(dy),
        reinterpret_cast<half*>(dx));
  }
  static void TanH(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
    TanHForwardGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, reinterpret_cast<const half*>(x), reinterpret_cast<half*>(y));
  }
  static void TanHBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, const T* dy,
                           T* dx) {
    TanHBackwardGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, reinterpret_cast<const half*>(y), reinterpret_cast<const half*>(dy),
        reinterpret_cast<half*>(dx));
  }
  static void Relu(DeviceCtx* ctx, const int64_t n, const T* x, T* y) {
    ReluForwardGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, reinterpret_cast<const half*>(x), reinterpret_cast<half*>(y));
  }
  static void ReluBackward(DeviceCtx* ctx, const int64_t n, const T* x, const T* y, const T* dy,
                           T* dx) {
    ReluBackwardGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, reinterpret_cast<const half*>(y), reinterpret_cast<const half*>(dy),
        reinterpret_cast<half*>(dx));
  }
  static void Set(DeviceCtx* ctx, const T value, T* addr) {
    gpu_set<half>
        <<<1, 1, 0, ctx->hip_stream()>>>(static_cast<half>(value), reinterpret_cast<half*>(addr));
  }
  static void Axpy(DeviceCtx* ctx, const int n, const T alpha, const T* x, const int incx, T* y,
                   const int incy) {
    // half ha;
    // ha.setx(alpha.getx());
    AxpyHalfGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, float16_2half(alpha), reinterpret_cast<const half*>(x), incx, reinterpret_cast<half*>(y),
        incy);
  }
  static void Scal(DeviceCtx* ctx, const int n, const T alpha, T* x, const int incx) {
    // half ha;
    // ha.setx(alpha.getx());
    ScalHalfGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, float16_2half(alpha), reinterpret_cast<half*>(x), incx);
  }
};

template<typename T>
struct FloatingNewKernelUtilIf<DeviceType::kGPU, T> {
  static void Gemm(DeviceCtx* ctx, const enum CBLAS_ORDER order, const enum CBLAS_TRANSPOSE trans_a,
                   const enum CBLAS_TRANSPOSE trans_b, const int m, const int n, const int k,
                   const T alpha, const T* a, const int lda, const T* b, const int ldb,
                   const T beta, T* c,
                   const int ldc) {  // TODO: wrong HIPBLAS_OP_N
    hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
    hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
    cublas_gemm<T>(ctx->cublas_pmh_handle(), cublas_trans_b, cublas_trans_a, n, m, k, &alpha, b,
                   ldb, a, lda, &beta, c, ldc);
  }
};

template<typename T>
struct Float16NewKernelUtilIf<DeviceType::kGPU, T> {
  static void HGemm(DeviceCtx* ctx, const enum CBLAS_ORDER order,
                    const enum CBLAS_TRANSPOSE trans_a, const enum CBLAS_TRANSPOSE trans_b,
                    const int m, const int n, const int k, const T alpha, const T* a, const int lda,
                    const T* b, const int ldb, const T beta, T* c, const int ldc) {
    hipblasOperation_t cublas_trans_a = CblasTrans2CublasTrans(trans_a);
    hipblasOperation_t cublas_trans_b = CblasTrans2CublasTrans(trans_b);
    CudaCheck(hipblasHgemm(ctx->cublas_tensor_op_math_handle(), cublas_trans_b, cublas_trans_a, n, m,
                          k, reinterpret_cast<const half*>(&alpha),
                          reinterpret_cast<const half*>(b), ldb, reinterpret_cast<const half*>(a),
                          lda, reinterpret_cast<const half*>(&beta), reinterpret_cast<half*>(c),
                          ldc));
  }
  static void Half2Float(DeviceCtx* ctx, const int n, const T* src, float* dst) {
    Half2FloatGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, reinterpret_cast<const half*>(src), dst);
  }
  static void Float2Half(DeviceCtx* ctx, const int n, const float* src, T* dst) {
    Float2HalfGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, src, reinterpret_cast<half*>(dst));
  }
};

#define INSTANTIATE_KERNEL_UTIL(type_cpp, type_proto) \
  template struct NewKernelUtilIf<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KERNEL_UTIL, ARITHMETIC_DATA_TYPE_SEQ FLOAT16_DATA_TYPE_SEQ);

#define INSTANTIATE_FLOATING_KERNEL_UTIL(type_cpp, type_proto) \
  template struct FloatingNewKernelUtilIf<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_FLOATING_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ);

#define INSTANTIATE_FLOAT16_KERNEL_UTIL(type_cpp, type_proto) \
  template struct Float16NewKernelUtilIf<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_FLOAT16_KERNEL_UTIL, FLOAT16_DATA_TYPE_SEQ);

}  // namespace oneflow
