#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/device/cuda_util.h"

namespace oneflow {

template<>
void Memcpy<DeviceType::kGPU>(DeviceCtx* ctx, void* dst, const void* src, size_t sz,
                              hipMemcpyKind kind) {
  if (dst == src) { return; }
  CudaCheck(hipMemcpyAsync(dst, src, sz, kind, ctx->hip_stream()));
}

}  // namespace oneflow
