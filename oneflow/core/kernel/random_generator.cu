#include "oneflow/core/kernel/random_generator.h"

namespace oneflow {

namespace {

template<typename T>
void RngUniformGpu(const hiprandGenerator_t& gen, int64_t n, T* ret);

template<>
void RngUniformGpu<float>(const hiprandGenerator_t& gen, int64_t n, float* ret) {
  CudaCheck(hiprandGenerateUniform(gen, ret, n));
}

template<>
void RngUniformGpu<double>(const hiprandGenerator_t& gen, int64_t n, double* ret) {
  CudaCheck(hiprandGenerateUniformDouble(gen, ret, n));
}

}  // namespace

RandomGeneratorGpuImpl::RandomGeneratorGpuImpl(int64_t seed, hipStream_t hip_stream) {
  CudaCheck(hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CudaCheck(hiprandSetPseudoRandomGeneratorSeed(curand_generator_, seed));
  CudaCheck(hiprandSetStream(curand_generator_, hip_stream));
}

RandomGeneratorGpuImpl::~RandomGeneratorGpuImpl() {
  CudaCheck(hiprandDestroyGenerator(curand_generator_));
}

template<typename T>
void RandomGeneratorGpuImpl::TUniform(const int64_t elem_cnt, T* dptr) {
  RngUniformGpu(curand_generator_, elem_cnt, dptr);
}

template void RandomGeneratorGpuImpl::TUniform<float>(const int64_t elem_cnt, float* dptr);
template void RandomGeneratorGpuImpl::TUniform<double>(const int64_t elem_cnt, double* dptr);

}  // namespace oneflow
