#include "oneflow/core/kernel/random_generator.h"

namespace oneflow {

namespace {

template<typename T>
void RngUniformGpu(const hiprandGenerator_t& gen, int64_t n, T* ret);

template<>
void RngUniformGpu<float>(const hiprandGenerator_t& gen, int64_t n, float* ret) {
  CudaCheck(hiprandGenerateUniform(gen, ret, n));
}

template<>
void RngUniformGpu<double>(const hiprandGenerator_t& gen, int64_t n, double* ret) {
  CudaCheck(hiprandGenerateUniformDouble(gen, ret, n));
}

}  // namespace

RandomGenerator<DeviceType::kGPU>::RandomGenerator(int64_t seed, DeviceCtx* device_ctx) {
  CHECK_NOTNULL(device_ctx);
  CudaCheck(hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CudaCheck(hiprandSetPseudoRandomGeneratorSeed(curand_generator_, seed));
  CudaCheck(hiprandSetStream(curand_generator_, device_ctx->hip_stream()));
}

RandomGenerator<DeviceType::kGPU>::~RandomGenerator() {
  CudaCheck(hiprandDestroyGenerator(curand_generator_));
}

template<typename T>
void RandomGenerator<DeviceType::kGPU>::Uniform(const int64_t elem_cnt, T* dptr) {
  RngUniformGpu(curand_generator_, elem_cnt, dptr);
}

#define INITIATE_GPU_RANDOM_GENERATOR_UNIFORM(T, typeproto) \
  template void RandomGenerator<DeviceType::kGPU>::Uniform<T>(const int64_t elem_cnt, T* dptr);

OF_PP_FOR_EACH_TUPLE(INITIATE_GPU_RANDOM_GENERATOR_UNIFORM, FLOATING_DATA_TYPE_SEQ);

}  // namespace oneflow
