/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/kernel/random_generator.h"

namespace oneflow {

namespace {

template<typename T>
void RngUniformGpu(const hiprandGenerator_t& gen, int64_t n, T* ret);

template<>
void RngUniformGpu<float>(const hiprandGenerator_t& gen, int64_t n, float* ret) {
  OF_CURAND_CHECK(hiprandGenerateUniform(gen, ret, n));
}

template<>
void RngUniformGpu<double>(const hiprandGenerator_t& gen, int64_t n, double* ret) {
  OF_CURAND_CHECK(hiprandGenerateUniformDouble(gen, ret, n));
}

}  // namespace

RandomGenerator<DeviceType::kGPU>::RandomGenerator(int64_t seed, DeviceCtx* device_ctx) {
  CHECK_NOTNULL(device_ctx);
  OF_CURAND_CHECK(hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
  OF_CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(curand_generator_, seed));
  OF_CURAND_CHECK(hiprandSetStream(curand_generator_, device_ctx->hip_stream()));
}

RandomGenerator<DeviceType::kGPU>::~RandomGenerator() {
  OF_CURAND_CHECK(hiprandDestroyGenerator(curand_generator_));
}

template<typename T>
void RandomGenerator<DeviceType::kGPU>::Uniform(const int64_t elem_cnt, T* dptr) {
  RngUniformGpu(curand_generator_, elem_cnt, dptr);
}

#define INITIATE_GPU_RANDOM_GENERATOR_UNIFORM(T, typeproto) \
  template void RandomGenerator<DeviceType::kGPU>::Uniform<T>(const int64_t elem_cnt, T* dptr);

OF_PP_FOR_EACH_TUPLE(INITIATE_GPU_RANDOM_GENERATOR_UNIFORM, FLOATING_DATA_TYPE_SEQ);

}  // namespace oneflow
