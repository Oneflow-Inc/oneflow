#include "oneflow/core/kernel/random_generator.h"

namespace oneflow {

namespace {

template<typename T>
void RngUniformGpu(const hiprandGenerator_t& gen, int64_t n, T* ret);

template<>
void RngUniformGpu<float>(const hiprandGenerator_t& gen, int64_t n, float* ret) {
  CudaCheck(hiprandGenerateUniform(gen, ret, n));
}

template<>
void RngUniformGpu<double>(const hiprandGenerator_t& gen, int64_t n, double* ret) {
  CudaCheck(hiprandGenerateUniformDouble(gen, ret, n));
}

}  // namespace

RandomGenerator<DeviceType::kGPU>::RandomGenerator(int64_t seed, hipStream_t hip_stream) {
  CudaCheck(hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CudaCheck(hiprandSetPseudoRandomGeneratorSeed(curand_generator_, seed));
  CudaCheck(hiprandSetStream(curand_generator_, hip_stream));
}

RandomGenerator<DeviceType::kGPU>::~RandomGenerator() {
  CudaCheck(hiprandDestroyGenerator(curand_generator_));
}

template<typename T>
void RandomGenerator<DeviceType::kGPU>::Uniform(const int64_t elem_cnt, T* dptr) {
  RngUniformGpu(curand_generator_, elem_cnt, dptr);
}

template void RandomGenerator<DeviceType::kGPU>::Uniform<float>(const int64_t elem_cnt,
                                                                float* dptr);
template void RandomGenerator<DeviceType::kGPU>::Uniform<double>(const int64_t elem_cnt,
                                                                 double* dptr);

}  // namespace oneflow
