#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/roi_pooling_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void RoIPoolForward(const int64_t nthreads, const T* in_dptr, const float spatial_scale,
                               const int64_t channel_num, const int64_t height, const int64_t width,
                               const int64_t roi_num, const int64_t pooled_height,
                               const int64_t pooled_width, const T* rois_dptr, T* out_dptr,
                               int32_t* argmax_dptr) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t pooled_area = pooled_width * pooled_height;
    const int64_t w = index % pooled_width;
    const int64_t h = (index / pooled_width) % pooled_height;
    const int64_t c = (index / pooled_area) % channel_num;
    const int64_t r = (index / pooled_area / channel_num) % roi_num;
    const int64_t n = index / pooled_area / channel_num / roi_num;
    const T* offset_rois = rois_dptr + (n * roi_num + r) * 4;
    int64_t roi_start_w =
        min(max(static_cast<int64_t>(round(offset_rois[0] * spatial_scale)), 0l), height);
    int64_t roi_start_h =
        min(max(static_cast<int64_t>(round(offset_rois[1] * spatial_scale)), 0l), width);
    int64_t roi_end_w =
        min(max(static_cast<int64_t>(round(offset_rois[2] * spatial_scale)), 0l), height);
    int64_t roi_end_h =
        min(max(static_cast<int64_t>(round(offset_rois[3] * spatial_scale)), 0l), width);
    int64_t roi_height = max(roi_end_h - roi_start_h + 1, 1l);
    int64_t roi_width = max(roi_end_w - roi_start_w + 1, 1l);
    const float bin_height = static_cast<float>(roi_height) / static_cast<float>(pooled_height);
    const float bin_width = static_cast<float>(roi_width) / static_cast<float>(pooled_width);
    int64_t hstart = floor(static_cast<float>(h) * bin_height);
    int64_t wstart = floor(static_cast<float>(w) * bin_width);
    int64_t hend = ceil(static_cast<float>(h + 1) * bin_height);
    int64_t wend = ceil(static_cast<float>(w + 1) * bin_width);
    hstart = min(max(roi_start_h + hstart, 0l), height);
    wstart = min(max(roi_start_w + wstart, 0l), width);
    hend = min(max(roi_start_h + hend, 0l), height);
    wend = min(max(roi_start_w + wend, 0l), width);
    bool is_bin_empty = (hend <= hstart) || (wend <= wstart);
    T max_val = is_bin_empty ? 0 : -FLT_MAX;
    int32_t max_idx = -1;
    if (!is_bin_empty) {
      const T* offset_in_dptr = in_dptr + (n * channel_num + c) * height * width;
      FOR_RANGE(int64_t, feat_h, hstart, hend) {
        FOR_RANGE(int64_t, feat_w, wstart, wend) {
          int32_t idx = feat_h * width + feat_w;
          if (offset_in_dptr[idx] > max_val) {
            max_val = offset_in_dptr[idx];
            max_idx = idx;
          }
        }
      }
    }
    out_dptr[index] = max_val;
    argmax_dptr[index] = max_idx;
  }
}

}  // namespace

template<typename T>
class RoIPoolingKernelUtil<DeviceType::kGPU, T> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(RoIPoolingKernelUtil);
  RoIPoolingKernelUtil() = delete;

  static void Forward(const KernelCtx& ctx, const RoIPoolingOpConf& conf, const Blob* in_blob,
                      const Blob* rois_blob, Blob* out_blob, Blob* argmax_blob) {
    const int64_t count = out_blob->shape().elem_cnt();
    RoIPoolForward<T><<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
                        ctx.device_ctx->hip_stream()>>>(
        count, in_blob->dptr<T>(), conf.spatial_scale(), in_blob->shape().At(1),
        in_blob->shape().At(2), in_blob->shape().At(3), rois_blob->shape().At(1), conf.pooled_h(),
        conf.pooled_w(), rois_blob->dptr<T>(), out_blob->mut_dptr<T>(),
        argmax_blob->mut_dptr<int32_t>());
  }

  static void Backward(const KernelCtx& ctx, const RoIPoolingOpConf& conf,
                       const Blob* out_diff_blob, const Blob* rois_blob, const Blob* argmax_blob,
                       Blob* in_diff_blob) {}
};

#define INSTANTIATE_ROI_POOLING_KERNEL_UTIL(type_cpp, type_proto) \
  template class RoIPoolingKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_ROI_POOLING_KERNEL_UTIL,
                     OF_PP_MAKE_TUPLE_SEQ(float, DataType::kFloat))

}  // namespace oneflow
