#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/smooth_l1_loss_kernel.h"

namespace oneflow {

namespace {

template<typename PredType, typename LabelType>
__global__ void SmoothL1LossForward(const int64_t N, const int64_t D, const PredType* prediction,
                                    const LabelType* label, const int8_t* inside_weights,
                                    const int8_t* outside_weights, const float beta,
                                    const float scale, PredType* loss_buf) {
  int64_t elem_cnt = N * D;
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    PredType x = inside_weights[i] * (prediction[i] - label[i]);
    PredType abs_x = abs(x);
    if (abs_x < beta) {
      loss_buf[i] = 0.5 * x * x / beta;
    } else {
      loss_buf[i] = abs_x - 0.5 * beta;
    }
    loss_buf[i] *= scale / elem_cnt * outside_weights[i];
  }
}

template<typename PredType, typename LabelType>
__global__ void SmoothL1LossBackward(const int64_t N, const int64_t D, const PredType* prediction,
                                     const LabelType* label, const int8_t* inside_weights,
                                     const int8_t* outside_weights, const float beta,
                                     const float scale, PredType* in_diff) {
  int64_t elem_cnt = N * D;
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    PredType x = inside_weights[i] * (prediction[i] - label[i]);
    PredType abs_x = abs(x);
    if (abs_x < beta) {
      in_diff[i] = x / beta;
    } else {
      in_diff[i] = x > 0 ? 1 : -1;
    }
    in_diff[i] *= scale / elem_cnt * outside_weights[i];
  }
}

}  // namespace

template<typename PredType, typename LabelType>
struct SmoothL1LossKernelUtil<DeviceType::kGPU, PredType, LabelType> {
  static void Forward(DeviceCtx* ctx, const int64_t N, const int64_t D, const PredType* prediction,
                      const LabelType* label, const int8_t* inside_weights,
                      const int8_t* outside_weights, const PredType* const_all_one,
                      const float beta, const float scale, PredType* loss_buf, PredType* loss) {
    SmoothL1LossForward<PredType>
        <<<BlocksNum4ThreadsNum(N * D), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            N, D, prediction, label, inside_weights, outside_weights, beta, scale, loss_buf);
    KernelUtil<DeviceType::kGPU, PredType>::Dot(ctx, N * D, loss_buf, 1, const_all_one, 1, loss);
  }
  static void Backward(DeviceCtx* ctx, const int64_t N, const int64_t D, const PredType* prediction,
                       const LabelType* label, const int8_t* inside_weights,
                       const int8_t* outside_weights, const float beta, const float scale,
                       PredType* in_diff) {
    SmoothL1LossBackward<PredType>
        <<<BlocksNum4ThreadsNum(N * D), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            N, D, prediction, label, inside_weights, outside_weights, beta, scale, in_diff);
  }
};

#define MAKE_ENTRY(data_type_pair, label_type_pair)                                          \
  template struct SmoothL1LossKernelUtil<DeviceType::kGPU, OF_PP_PAIR_FIRST(data_type_pair), \
                                         OF_PP_PAIR_FIRST(label_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_ENTRY, FLOATING_DATA_TYPE_SEQ, INT_DATA_TYPE_SEQ)
}  // namespace oneflow
