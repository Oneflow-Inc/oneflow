#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/momentum_model_update_kernel.h"
#include "oneflow/core/kernel/normal_model_update_kernel.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void UpdateModelGpu(int64_t n, int64_t batch_size, T beta, T learning_rate, T l1, T l2,
                               const T* model_diff, const T* pre_model, T* momentum, T* model) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    T reg_diff = RegularizeDiff(model_diff[i], batch_size, l1, l2, pre_model[i]);
    momentum[i] = beta * momentum[i] - learning_rate * reg_diff;
    model[i] = pre_model[i] + momentum[i];
  }
}

}  // namespace

template<typename T>
class MomentumMdUpdateKernelUtil<DeviceType::kGPU, T> final {
 public:
  static void UpdateModel(DeviceCtx* ctx, int64_t n, int64_t batch_size, T beta, T learning_rate,
                          const T l1, const T l2, const T* model_diff, const T* pre_model,
                          T* momentum, T* model) {
    UpdateModelGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, batch_size, beta, learning_rate, l1, l2, model_diff, pre_model, momentum, model);
  }
};

#define INSTANTIATE_GPU_KERNEL_UTIL(type_cpp, type_proto) \
  template class MomentumMdUpdateKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GPU_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
