#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/momentum_model_update_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void UpdateModelGpu(int64_t n, int64_t batch_size, T beta, T learning_rate, T l1, T l2,
                               const T* model_diff, const T* pre_model, T* momentum, T* model) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    T avg_model_diff = model_diff[i] / batch_size;
    momentum[i] = beta * momentum[i] - learning_rate * (avg_model_diff + l2 * pre_model[i]);
    model[i] = pre_model[i] + momentum[i];
    model[i] -= l1 * ((pre_model[i] >= 0) - (pre_model[i] <= 0));
  }
}

}  // namespace

template<typename T>
class MomentumMdUpdateKernelUtil<DeviceType::kGPU, T> final {
 public:
  static void UpdateModel(DeviceCtx* ctx, int64_t n, int64_t batch_size, T beta, T learning_rate,
                          const T l1, const T l2, const T* model_diff, const T* pre_model,
                          T* momentum, T* model) {
    UpdateModelGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, batch_size, beta, learning_rate, l1, l2, model_diff, pre_model, momentum, model);
  }
};

#define INSTANTIATE_GPU_KERNEL_UTIL(type_cpp, type_proto) \
  template class MomentumMdUpdateKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GPU_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
