#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/kernel/random_generator.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void AtomicExchange(const int64_t n, T* in, const int64_t retry_limit) {
  CUDA_1D_KERNEL_LOOP(i, n) {}
}

}  // namespace

template<typename T>
class RandomPermLikeGPUKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(RandomPermLikeGPUKernel);
  RandomPermLikeGPUKernel() = default;
  ~RandomPermLikeGPUKernel() = default;

  void VirtualKernelInit(DeviceCtx* device_ctx) {
    const auto& dropout_conf = this->op_conf().dropout_conf();
    int64_t seed = -1;
    const RandomPermLikeOpConf& random_perm_like_conf = this->op_conf().random_perm_like_conf();
    if (random_perm_like_conf.has_random_seed()) {
      seed = random_perm_like_conf.random_seed();
    } else {
      seed = GetCurTime();
    }
    CHECK_NE(seed, -1);
    random_generator_.reset(new RandomGenerator<DeviceType::kGPU>(seed, device_ctx));
  }

  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    Blob* out_blob = BnInOp2Blob("out");
    Blob* random_mask_blob = BnInOp2Blob("random_mask");
    const int64_t n = out_blob->shape().At(0);
    random_generator_->Uniform(n, random_mask_blob->mut_dptr<float>());
  }

 private:
  std::unique_ptr<RandomGenerator<DeviceType::kGPU>> random_generator_;
};

#define REGISTER_RANDOM_PERM_GPU_KERNEL(dtype)                                               \
  REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kRandomPermLikeConf, DeviceType::kGPU, \
                                        dtype, RandomPermLikeGPUKernel<dtype>);

REGISTER_RANDOM_PERM_GPU_KERNEL(int32_t);

}  // namespace oneflow
