#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/kernel/check_numerics_kernel_observer.h"
#include "oneflow/core/kernel/kernel.h"

namespace oneflow {

namespace {

template<typename T>
__device__ bool IsNotFinite(T x) {
  return !isfinite(x);
}

template<>
__device__ bool IsNotFinite<half>(half x) {
  return (__hisinf(x) || __hisnan(x));
}

template<typename T>
__global__ void HasNotFiniteGpuKernel(const int64_t n, const T* x, volatile bool* has_not_finite) {
  if (*has_not_finite) { return; }
  CUDA_1D_KERNEL_LOOP_T(int64_t, i, n) {
    if (IsNotFinite(x[i])) {
      *has_not_finite = true;
      return;
    }
  }
}

class HasNotFiniteContext {
 public:
  OF_DISALLOW_COPY_AND_MOVE(HasNotFiniteContext);
  HasNotFiniteContext() = default;
  ~HasNotFiniteContext() {
    if (has_not_finite_host_ != nullptr) { OF_CUDA_CHECK(hipHostFree(has_not_finite_host_)); }
    if (has_not_finite_device_ != nullptr) { OF_CUDA_CHECK(hipFree(has_not_finite_device_)); }
  }

  bool* has_not_finite_host() {
    if (has_not_finite_host_ == nullptr) {
      OF_CUDA_CHECK(hipHostMalloc(&has_not_finite_host_, sizeof(bool)));
    }
    return has_not_finite_host_;
  }
  bool* has_not_finite_device() {
    if (has_not_finite_device_ == nullptr) {
      OF_CUDA_CHECK(hipMalloc(&has_not_finite_device_, sizeof(bool)));
    }
    return has_not_finite_device_;
  }

 private:
  bool* has_not_finite_host_;
  bool* has_not_finite_device_;
};

template<typename T>
bool HasNotFinite(DeviceCtx* device_ctx, const int64_t elem_cnt, const T* data_ptr) {
  thread_local static HasNotFiniteContext ctx;
  OF_CUDA_CHECK(
      hipMemsetAsync(ctx.has_not_finite_device(), 0, sizeof(bool), device_ctx->hip_stream()));
  HasNotFiniteGpuKernel<T>
      <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, device_ctx->hip_stream()>>>(
          elem_cnt, data_ptr, ctx.has_not_finite_device());
  OF_CUDA_CHECK(hipMemcpyAsync(ctx.has_not_finite_host(), ctx.has_not_finite_device(),
                                sizeof(bool), hipMemcpyDefault, device_ctx->hip_stream()));
  OF_CUDA_CHECK(hipStreamSynchronize(device_ctx->hip_stream()));
  return *ctx.has_not_finite_host();
}

}  // namespace

bool HasNotFiniteGpu(DeviceCtx* device_ctx, const Blob* blob) {
  const DataType dtype = blob->data_type();
  const int64_t elem_cnt = blob->shape().elem_cnt();
  if (dtype == kFloat) {
    return HasNotFinite<float>(device_ctx, elem_cnt, blob->dptr<float>());
  } else if (dtype == kDouble) {
    return HasNotFinite<double>(device_ctx, elem_cnt, blob->dptr<double>());
  } else if (dtype == kFloat16) {
    return HasNotFinite<half>(device_ctx, elem_cnt, blob->dptr<half>());
  } else {
    return false;
  }
}

}  // namespace oneflow
