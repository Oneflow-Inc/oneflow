#include "oneflow/core/kernel/conv_kernel.h"
#include "oneflow/core/kernel/kernel_util.h"

namespace oneflow {

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::VirtualKernelInit(
    const ParallelContext* parallel_ctx) {
  Shape in_shape(static_cast<const ShapeProto&>(
      this->GetMessageFromCustomizedKernelConf("in")));
  Shape out_shape(static_cast<const ShapeProto&>(
      this->GetMessageFromCustomizedKernelConf("out")));
  Shape weight_shape(static_cast<const ShapeProto&>(
      this->GetMessageFromCustomizedKernelConf("weight")));

  std::vector<int32_t> stride_of_in_tensor(this->KernelDim(), 1);
  std::vector<int32_t> stride_of_out_tensor(this->KernelDim(), 1);
  for (int32_t i = this->KernelDim() + 2 - 1; i > 0; --i) {
    for (int32_t j = this->KernelDim() + 2 - 2; j >= 0; --j) {
      stride_of_in_tensor[j] *= in_shape.At(i);
      stride_of_out_tensor[j] *= out_shape.At(i);
    }
  }
  std::vector<int32_t> in_dim(in_shape.dim_vec().begin(),
                              in_shape.dim_vec().end());
  std::vector<int32_t> out_dim(out_shape.dim_vec().begin(),
                               out_shape.dim_vec().end());

  this->in_desc_.reset(new CudnnTensorDesc(GetDataType<T>::val,
                                           this->KernelDim() + 2, in_dim.data(),
                                           stride_of_in_tensor.data()));
  this->out_desc_.reset(
      new CudnnTensorDesc(GetDataType<T>::val, this->KernelDim() + 2,
                          out_dim.data(), stride_of_out_tensor.data()));
  this->filter_desc_.reset(
      new CudnnFilterDesc(GetDataType<T>::val, weight_shape,
                          this->GetStringFromCustomizedOpConf("data_format")));
  this->conv_desc_.reset(new CudnnConvDesc(GetDataType<T>::val, in_shape,
                                           this->GetCustomizedOpConf()));

  if (this->GetBoolFromCustomizedOpConf("use_bias")) {
    int32_t filters = this->GetInt32FromCustomizedOpConf("filters");
    int32_t stride_of_bias_tensor = 1;
    this->bias_desc_.reset(new CudnnTensorDesc(GetDataType<T>::val, 1, &filters,
                                               &stride_of_bias_tensor));
  }
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::WeightForward(
    DeviceCtx* device_ctx, const Blob* in, const Blob* weight, Blob* out,
    std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  Blob* cudnn_buf = BnInOp2Blob("cudnn_buf");
  CudaCheck(hipdnnConvolutionForward(
      device_ctx->cudnn_handle(), CudnnDataType<T>::one, this->in_desc_->Get(),
      in->dptr<T>(), this->filter_desc_->Get(), weight->dptr<T>(),
      this->conv_desc_->Get(),
      static_cast<hipdnnConvolutionFwdAlgo_t>(
          this->GetInt32FromCustomizedKernelConf("cudnn_fwd_algo")),
      cudnn_buf->mut_dptr<T>(), cudnn_buf->shape().At(0),
      CudnnDataType<T>::zero, this->out_desc_->Get(), out->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::BiasForward(DeviceCtx* device_ctx,
                                                  const Blob* bias,
                                                  Blob* out) const {
  CudaCheck(hipdnnAddTensor(device_ctx->cudnn_handle(), CudnnDataType<T>::one,
                           this->bias_desc_->Get(), bias->dptr<T>(),
                           CudnnDataType<T>::one, this->out_desc_->Get(),
                           out->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::DataBackward(
    DeviceCtx* device_ctx, const Blob* out_diff, const Blob* weight,
    Blob* in_diff, std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  Blob* cudnn_buf = BnInOp2Blob("cudnn_buf");
  CudaCheck(hipdnnConvolutionBackwardData(
      device_ctx->cudnn_handle(), CudnnDataType<T>::one,
      this->filter_desc_->Get(), weight->dptr<T>(), this->out_desc_->Get(),
      out_diff->dptr<T>(), this->conv_desc_->Get(),
      static_cast<hipdnnConvolutionBwdDataAlgo_t>(
          this->GetInt32FromCustomizedKernelConf("cudnn_bwd_data_algo")),
      cudnn_buf->mut_dptr<T>(), cudnn_buf->shape().At(0),
      CudnnDataType<T>::zero, this->in_desc_->Get(), in_diff->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::WeightBackward(
    DeviceCtx* device_ctx, const Blob* out_diff, const Blob* in,
    Blob* weight_diff,
    std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  Blob* cudnn_buf = BnInOp2Blob("cudnn_buf");
  CudaCheck(hipdnnConvolutionBackwardFilter(
      device_ctx->cudnn_handle(), CudnnDataType<T>::one, this->in_desc_->Get(),
      in->dptr<T>(), this->out_desc_->Get(), out_diff->dptr<T>(),
      this->conv_desc_->Get(),
      static_cast<hipdnnConvolutionBwdFilterAlgo_t>(
          this->GetInt32FromCustomizedKernelConf("cudnn_bwd_filter_algo")),
      cudnn_buf->mut_dptr<T>(), cudnn_buf->shape().At(0), CudnnDataType<T>::one,
      this->filter_desc_->Get(), weight_diff->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::BiasBackward(DeviceCtx* device_ctx,
                                                   const Blob* out_diff,
                                                   Blob* bias_diff) const {
  CudaCheck(hipdnnConvolutionBackwardBias(
      device_ctx->cudnn_handle(), CudnnDataType<T>::one, this->out_desc_->Get(),
      out_diff->dptr<T>(), CudnnDataType<T>::one, this->bias_desc_->Get(),
      bias_diff->mut_dptr<T>()));
}

#define INSTANTIATE_CONV_KERNEL(type_cpp, type_proto) \
  template class ConvKernel<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_CONV_KERNEL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
