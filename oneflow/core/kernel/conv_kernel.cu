#include "oneflow/core/kernel/conv_kernel.h"
#include "oneflow/core/kernel/kernel_util.h"

namespace oneflow {

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::VirtualKernelInit(
    const ParallelContext* parallel_ctx) {
  Shape in_shape(this->GetConvKernelConf().in());
  Shape out_shape(this->GetConvKernelConf().out());
  Shape weight_shape(this->GetConvKernelConf().weight());

  this->in_desc_.reset(
      new CudnnTensorDesc(GetDataType<T>::value, in_shape,
                          this->GetStringFromCustomizedOpConf("data_format")));
  this->out_desc_.reset(
      new CudnnTensorDesc(GetDataType<T>::value, out_shape,
                          this->GetStringFromCustomizedOpConf("data_format")));
  this->filter_desc_.reset(
      new CudnnFilterDesc(GetDataType<T>::value, weight_shape,
                          this->GetStringFromCustomizedOpConf("data_format")));
  this->conv_desc_.reset(new CudnnConvDesc(GetDataType<T>::value, in_shape,
                                           this->GetCustomizedOpConf()));

  if (this->GetBoolFromCustomizedOpConf("use_bias")) {
    int32_t filters = this->GetInt32FromCustomizedOpConf("filters");
    std::vector<int32_t> bias_dim(this->OpKernelDim() + 2, 1);
    std::vector<int32_t> stride_of_bias_tensor(this->OpKernelDim() + 2, 1);
    bias_dim[1] = filters;
    stride_of_bias_tensor[0] = filters;

    this->bias_desc_.reset(
        new CudnnTensorDesc(GetDataType<T>::value, this->OpKernelDim() + 2,
                            bias_dim.data(), stride_of_bias_tensor.data()));
  }
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::WeightForward(
    DeviceCtx* device_ctx, const Blob* in, const Blob* weight, Blob* out,
    std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  Blob* cudnn_buf = BnInOp2Blob("cudnn_buf");
  int64_t cudnn_buf_size = 0;
  T* cudnn_buf_dptr = nullptr;
  if (cudnn_buf) {
    cudnn_buf_size = cudnn_buf->shape().At(0);
    cudnn_buf_dptr = cudnn_buf->mut_dptr<T>();
  }
  CudaCheck(hipdnnConvolutionForward(
      device_ctx->cudnn_handle(), OnePtr<T>::value, this->in_desc_->Get(),
      in->dptr<T>(), this->filter_desc_->Get(), weight->dptr<T>(),
      this->conv_desc_->Get(),
      static_cast<hipdnnConvolutionFwdAlgo_t>(
          this->GetConvKernelConf().cudnn_fwd_algo()),
      cudnn_buf_dptr, cudnn_buf_size, ZeroPtr<T>::value, this->out_desc_->Get(),
      out->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::BiasForward(DeviceCtx* device_ctx,
                                                  const Blob* bias,
                                                  Blob* out) const {
  CudaCheck(hipdnnAddTensor(device_ctx->cudnn_handle(), OnePtr<T>::value,
                           this->bias_desc_->Get(), bias->dptr<T>(),
                           OnePtr<T>::value, this->out_desc_->Get(),
                           out->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::DataBackward(
    DeviceCtx* device_ctx, const Blob* out_diff, const Blob* weight,
    Blob* in_diff, std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  Blob* cudnn_buf = BnInOp2Blob("cudnn_buf");
  int64_t cudnn_buf_size = 0;
  T* cudnn_buf_dptr = nullptr;
  if (cudnn_buf) {
    cudnn_buf_size = cudnn_buf->shape().At(0);
    cudnn_buf_dptr = cudnn_buf->mut_dptr<T>();
  }
  CudaCheck(hipdnnConvolutionBackwardData(
      device_ctx->cudnn_handle(), OnePtr<T>::value, this->filter_desc_->Get(),
      weight->dptr<T>(), this->out_desc_->Get(), out_diff->dptr<T>(),
      this->conv_desc_->Get(),
      static_cast<hipdnnConvolutionBwdDataAlgo_t>(
          this->GetConvKernelConf().cudnn_bwd_data_algo()),
      cudnn_buf_dptr, cudnn_buf_size, ZeroPtr<T>::value, this->in_desc_->Get(),
      in_diff->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::WeightBackward(
    DeviceCtx* device_ctx, const Blob* out_diff, const Blob* in,
    Blob* weight_diff,
    std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  Blob* cudnn_buf = BnInOp2Blob("cudnn_buf");
  int64_t cudnn_buf_size = 0;
  T* cudnn_buf_dptr = nullptr;
  if (cudnn_buf) {
    cudnn_buf_size = cudnn_buf->shape().At(0);
    cudnn_buf_dptr = cudnn_buf->mut_dptr<T>();
  }
  CudaCheck(hipdnnConvolutionBackwardFilter(
      device_ctx->cudnn_handle(), OnePtr<T>::value, this->in_desc_->Get(),
      in->dptr<T>(), this->out_desc_->Get(), out_diff->dptr<T>(),
      this->conv_desc_->Get(),
      static_cast<hipdnnConvolutionBwdFilterAlgo_t>(
          this->GetConvKernelConf().cudnn_bwd_filter_algo()),
      cudnn_buf_dptr, cudnn_buf_size, ZeroPtr<T>::value,
      this->filter_desc_->Get(), weight_diff->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::BiasBackward(DeviceCtx* device_ctx,
                                                   const Blob* out_diff,
                                                   Blob* bias_diff) const {
  CudaCheck(hipdnnConvolutionBackwardBias(
      device_ctx->cudnn_handle(), OnePtr<T>::value, this->out_desc_->Get(),
      out_diff->dptr<T>(), ZeroPtr<T>::value, this->bias_desc_->Get(),
      bias_diff->mut_dptr<T>()));
}

#define INSTANTIATE_CONV_KERNEL(type_cpp, type_proto) \
  template class ConvKernel<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_CONV_KERNEL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
