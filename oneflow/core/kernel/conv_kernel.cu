#include "oneflow/core/kernel/conv_kernel.h"
#include "oneflow/core/kernel/kernel_util.h"

namespace oneflow {

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::VirtualKernelInit(
    const ParallelContext* parallel_ctx) {
  Shape in_shape(static_cast<const ShapeProto&>(
      this->GetMessageFromCustomizedKernelConf("in")));
  Shape out_shape(static_cast<const ShapeProto&>(
      this->GetMessageFromCustomizedKernelConf("out")));
  Shape weight_shape(static_cast<const ShapeProto&>(
      this->GetMessageFromCustomizedKernelConf("weight")));

  std::vector<int32_t> stride_of_in_tensor(this->KernelDim(), 1);
  std::vector<int32_t> stride_of_out_tensor(this->KernelDim(), 1);
  for (int32_t i = this->KernelDim() + 2 - 1; i > 0; --i) {
    for (int32_t j = this->KernelDim() + 2 - 2; j >= 0; --j) {
      stride_of_in_tensor[j] *= in_shape.At(i);
      stride_of_out_tensor[j] *= out_shape.At(i);
    }
  }
  std::vector<int32_t> in_dim(in_shape.dim_vec().begin(),
                              in_shape.dim_vec().end());
  std::vector<int32_t> out_dim(out_shape.dim_vec().begin(),
                               out_shape.dim_vec().end());

  this->in_desc_.reset(new CudnnTensorDesc(GetDataType<T>::val,
                                           this->KernelDim() + 2, in_dim.data(),
                                           stride_of_in_tensor.data()));
  this->out_desc_.reset(
      new CudnnTensorDesc(GetDataType<T>::val, this->KernelDim() + 2,
                          out_dim.data(), stride_of_out_tensor.data()));
  this->filter_desc_.reset(
      new CudnnFilterDesc(GetDataType<T>::val, weight_shape,
                          this->GetStringFromCustomizedOpConf("data_format")));
  this->conv_desc_.reset(new CudnnConvDesc(
      GetDataType<T>::val, in_shape, this->KernelDim(),
      this->template GetPbRfFromCustomizedOpConf<int32_t>("dilation_rate")
          .data(),
      this->template GetPbRfFromCustomizedOpConf<int32_t>("strides").data(),
      this->template GetPbRfFromCustomizedOpConf<int32_t>("kernel_size").data(),
      this->GetStringFromCustomizedOpConf("data_format"),
      this->GetStringFromCustomizedOpConf("padding")));

  if (this->GetBoolFromCustomizedOpConf("use_bias")) {
    int32_t filters = this->GetInt32FromCustomizedOpConf("filters");
    int32_t stride_of_bias_tensor = 1;
    this->bias_desc_.reset(new CudnnTensorDesc(GetDataType<T>::val, 1, &filters,
                                               &stride_of_bias_tensor));
  }
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::WeightForward(
    DeviceCtx* device_ctx, const Blob* in_blob, const Blob* weight_blob,
    Blob* out_blob, Blob* cudnn_workspace) const {
  CudaCheck(hipdnnConvolutionForward(
      device_ctx->cudnn_handle(), CudnnDataType<T>::one, this->in_desc_->Get(),
      in_blob->dptr<T>(), this->filter_desc_->Get(), weight_blob->dptr<T>(),
      this->conv_desc_->Get(),
      static_cast<hipdnnConvolutionFwdAlgo_t>(
          this->GetInt32FromCustomizedKernelConf("cudnn_fwd_algo")),
      cudnn_workspace->mut_dptr<T>(), cudnn_workspace->shape().At(0),
      CudnnDataType<T>::zero, this->out_desc_->Get(), out_blob->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::BiasForward(DeviceCtx* device_ctx,
                                                  const Blob* bias_blob,
                                                  Blob* out_blob) const {
  CudaCheck(hipdnnAddTensor(device_ctx->cudnn_handle(), CudnnDataType<T>::one,
                           this->bias_desc_->Get(), bias_blob->dptr<T>(),
                           CudnnDataType<T>::one, this->out_desc_->Get(),
                           out_blob->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::DataBackward(
    DeviceCtx* device_ctx, const Blob* out_diff_blob, const Blob* weight_blob,
    Blob* in_diff_blob, Blob* cudnn_workspace) const {
  CudaCheck(hipdnnConvolutionBackwardData(
      device_ctx->cudnn_handle(), CudnnDataType<T>::one,
      this->filter_desc_->Get(), weight_blob->dptr<T>(), this->out_desc_->Get(),
      out_diff_blob->dptr<T>(), this->conv_desc_->Get(),
      static_cast<hipdnnConvolutionBwdDataAlgo_t>(
          this->GetInt32FromCustomizedKernelConf("cudnn_bwd_data_algo")),
      cudnn_workspace->mut_dptr<T>(), cudnn_workspace->shape().At(0),
      CudnnDataType<T>::zero, this->in_desc_->Get(),
      in_diff_blob->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::WeightBackward(
    DeviceCtx* device_ctx, const Blob* out_diff_blob, const Blob* in_blob,
    Blob* weight_diff_blob, Blob* cudnn_workspace) const {
  CudaCheck(hipdnnConvolutionBackwardFilter(
      device_ctx->cudnn_handle(), CudnnDataType<T>::one, this->in_desc_->Get(),
      in_blob->dptr<T>(), this->out_desc_->Get(), out_diff_blob->dptr<T>(),
      this->conv_desc_->Get(),
      static_cast<hipdnnConvolutionBwdFilterAlgo_t>(
          this->GetInt32FromCustomizedKernelConf("cudnn_bwd_filter_algo")),
      cudnn_workspace->mut_dptr<T>(), cudnn_workspace->shape().At(0),
      CudnnDataType<T>::one, this->filter_desc_->Get(),
      weight_diff_blob->mut_dptr<T>()));
}

template<typename T>
void ConvKernel<DeviceType::kGPU, T>::BiasBackward(DeviceCtx* device_ctx,
                                                   const Blob* out_diff_blob,
                                                   Blob* bias_diff_blob) const {
  CudaCheck(hipdnnConvolutionBackwardBias(
      device_ctx->cudnn_handle(), CudnnDataType<T>::one, this->out_desc_->Get(),
      out_diff_blob->dptr<T>(), CudnnDataType<T>::one, this->bias_desc_->Get(),
      bias_diff_blob->mut_dptr<T>()));
}

#define INSTANTIATE_CONV_KERNEL(type_cpp, type_proto) \
  template class ConvKernel<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_CONV_KERNEL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
