#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/device/cudnn_conv_util.h"

namespace oneflow {

template<typename T>
class ConvDataGradGpuKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(ConvDataGradGpuKernel);
  ConvDataGradGpuKernel() = default;
  ~ConvDataGradGpuKernel() = default;

 private:
  const PbMessage& GetCustomizedOpConf() const override {
    return this->op_conf().conv_data_grad_conf();
  }

  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    CudnnConvArgs args(this->job_desc().job_conf(),
                       this->op_conf().conv_data_grad_conf().conv_conf(),
                       ctx.device_ctx->cudnn_handle(), BnInOp2Blob("dx"), BnInOp2Blob("dy"),
                       BnInOp2Blob("filter"), BnInOp2Blob("buf"));
    auto algo_perf = FindCudnnConvAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>(&args);
    CHECK_EQ(algo_perf.status, HIPDNN_STATUS_SUCCESS);
    CHECK_LE(algo_perf.memory, BnInOp2Blob("buf")->ByteSizeOfBlobBody());
    CudaCheck(hipdnnConvolutionBackwardData(
        args.handle, CudnnSPOnePtr<T>(), args.wdesc.Get(), args.w_dptr, args.ydesc.Get(),
        args.y_dptr, args.cdesc.Get(), algo_perf.algo, args.ws_dptr, args.params.max_ws_size,
        CudnnSPZeroPtr<T>(), args.xdesc.Get(), args.x_dptr));
  }
};

#define REGISTER_CONV_DATA_GRAD_GPU_KERNEL(dtype)                                                 \
  REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kConvDataGradConf, DeviceType::kGPU, dtype, \
                                        ConvDataGradGpuKernel<dtype>);

REGISTER_CONV_DATA_GRAD_GPU_KERNEL(float);
REGISTER_CONV_DATA_GRAD_GPU_KERNEL(double);
REGISTER_CONV_DATA_GRAD_GPU_KERNEL(float16);

}  // namespace oneflow
