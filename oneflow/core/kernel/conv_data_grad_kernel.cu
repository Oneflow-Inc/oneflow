#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/device/cudnn_conv_util.h"

namespace oneflow {

template<typename T>
class ConvDataGradGpuKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(ConvDataGradGpuKernel);
  ConvDataGradGpuKernel() = default;
  ~ConvDataGradGpuKernel() = default;

 private:
  const PbMessage& GetCustomizedOpConf() const override {
    return this->op_conf().conv_data_grad_conf();
  }

  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    const Blob* dy = BnInOp2Blob("dy");
    const Blob* filter = BnInOp2Blob("filter");
    Blob* dx = BnInOp2Blob("dx");
    Blob* buf = BnInOp2Blob("buf");
    const ConvConf& conv_conf = this->op_conf().conv_data_grad_conf().conv_conf();
    CudnnConvArgs args(conv_conf, dx->data_type(), dx->shape(), filter->data_type(),
                       filter->shape(), dy->data_type(), dy->shape(), conv_conf.data_format(),
                       buf->ByteSizeOfBlobBody(),
                       this->job_desc().job_conf().cudnn_conv_heuristic_search_algo(),
                       this->job_desc().job_conf().cudnn_conv_use_deterministic_algo_only(),
                       this->job_desc().job_conf().cudnn_conv_enable_pseudo_half());
    AllocatedCudnnConvResource res(ctx.device_ctx->cudnn_handle(), dx->mut_dptr(),
                                   const_cast<void*>(filter->dptr()), const_cast<void*>(dy->dptr()),
                                   buf->mut_dptr());
    using perf_t = hipdnnConvolutionBwdDataAlgoPerf_t;
    using algo_t = hipdnnConvolutionBwdDataAlgo_t;
    perf_t algo_perf;
    if (this->job_desc().job_conf().has_cudnn_conv_force_bwd_data_algo()) {
      algo_perf = GetCudnnConvAlgorithmPerferenceWithResource<perf_t>(
          &args, &res,
          static_cast<algo_t>(this->job_desc().job_conf().cudnn_conv_force_bwd_data_algo()));
    } else {
      algo_perf = FindCudnnConvAlgorithmWithResource<perf_t>(&args, &res);
    }
    CHECK_EQ(algo_perf.status, HIPDNN_STATUS_SUCCESS)
        << "op (" << this->op_conf().name()
        << ") find algorithm perference failed. algo: " << algo_perf.algo;
    CHECK_LE(algo_perf.memory, buf->ByteSizeOfBlobBody())
        << "op (" << this->op_conf().name() << ") find algorithm " << algo_perf.algo
        << ", need memory " << algo_perf.memory << ", but cudnn_buf_limit_byte is "
        << buf->ByteSizeOfBlobBody();
    CudaCheck(hipdnnConvolutionBackwardData(
        ctx.device_ctx->cudnn_handle(), CudnnSPOnePtr<T>(), args.wdesc.Get(), filter->dptr(),
        args.ydesc.Get(), dy->dptr(), args.cdesc.Get(), algo_perf.algo, buf->mut_dptr(),
        args.params.max_ws_size, CudnnSPZeroPtr<T>(), args.xdesc.Get(), dx->mut_dptr()));
  }
};

#define REGISTER_CONV_DATA_GRAD_GPU_KERNEL(dtype)                                                 \
  REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kConvDataGradConf, DeviceType::kGPU, dtype, \
                                        ConvDataGradGpuKernel<dtype>);

REGISTER_CONV_DATA_GRAD_GPU_KERNEL(float);
REGISTER_CONV_DATA_GRAD_GPU_KERNEL(double);
REGISTER_CONV_DATA_GRAD_GPU_KERNEL(float16);

}  // namespace oneflow
