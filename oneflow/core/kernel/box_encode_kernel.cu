#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/box_encode_kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void EncodeGpu(const int32_t num_boxes, const T* ref_boxes_ptr, const T* boxes_ptr,
                          const float weight_x, const float weight_y, const float weight_w,
                          const float weight_h, T* boxes_delta_ptr) {
  const T TO_REMOVE = 1.0;
  CUDA_1D_KERNEL_LOOP(i, num_boxes) {
    const T ref_box_x1 = ref_boxes_ptr[i * 4];
    const T ref_box_y1 = ref_boxes_ptr[i * 4 + 1];
    const T ref_box_width = ref_boxes_ptr[i * 4 + 2] - ref_box_x1 + TO_REMOVE;
    const T ref_box_height = ref_boxes_ptr[i * 4 + 3] - ref_box_y1 + TO_REMOVE;
    const T box_x1 = boxes_ptr[i * 4];
    const T box_y1 = boxes_ptr[i * 4 + 1];
    const T box_width = boxes_ptr[i * 4 + 2] - box_x1 + TO_REMOVE;
    const T box_height = boxes_ptr[i * 4 + 3] - box_y1 + TO_REMOVE;
    boxes_delta_ptr[i * 4] =
        weight_x * ((ref_box_x1 + 0.5 * ref_box_width) - (box_x1 + 0.5 * box_width)) / box_width;
    boxes_delta_ptr[i * 4 + 1] =
        weight_y * ((ref_box_y1 + 0.5 * ref_box_height) - (box_y1 + 0.5 * box_height)) / box_height;
    boxes_delta_ptr[i * 4 + 2] = weight_w * log(ref_box_width / box_width);
    boxes_delta_ptr[i * 4 + 3] = weight_h * log(ref_box_height / box_height);
  }
}

}  // namespace

template<typename T>
struct BoxEncodeUtil<DeviceType::kGPU, T> {
  static void Encode(DeviceCtx* ctx, const int32_t num_boxes, const T* ref_boxes_ptr,
                     const T* boxes_ptr, const float weight_x, const float weight_y,
                     const float weight_w, const float weight_h, T* boxes_delta_ptr) {
    EncodeGpu<<<BlocksNum4ThreadsNum(num_boxes), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        num_boxes, ref_boxes_ptr, boxes_ptr, weight_x, weight_y, weight_w, weight_h,
        boxes_delta_ptr);
  }
};

#define MAKE_ENTRY(type_cpp, type_proto) template struct BoxEncodeUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(MAKE_ENTRY, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow