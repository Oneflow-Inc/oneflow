#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/kernel/cuda_check_numerics_kernel_observer.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

template<typename T>
__device__ bool IsNotFinite(T x) {
  return !isfinite(x);
}

#if __CUDA_ARCH__ >= 530
template<>
__device__ bool IsNotFinite<half>(half x) {
  return (__hisinf(x) || __hisnan(x));
}
#endif

template<typename T>
__global__ void HasNotFiniteGpuKernel(const int64_t n, const T* x, volatile bool* has_not_finite) {
  if (*has_not_finite) { return; }
  CUDA_1D_KERNEL_LOOP_T(int64_t, i, n) {
    if (IsNotFinite(x[i])) {
      *has_not_finite = true;
      return;
    }
  }
}

template<typename T>
bool HasNotFinite(ep::Stream* stream, const int64_t elem_cnt, const T* data_ptr,
                  bool* has_not_finite_host, bool* has_not_finite_device) {
  OF_CUDA_CHECK(hipMemsetAsync(has_not_finite_device, 0, sizeof(bool),
                                stream->As<ep::CudaStream>()->hip_stream()));
  HasNotFiniteGpuKernel<T>
      <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
         stream->As<ep::CudaStream>()->hip_stream()>>>(elem_cnt, data_ptr, has_not_finite_device);
  OF_CUDA_CHECK(hipMemcpyAsync(has_not_finite_host, has_not_finite_device, sizeof(bool),
                                hipMemcpyDefault, stream->As<ep::CudaStream>()->hip_stream()));
  OF_CUDA_CHECK(hipStreamSynchronize(stream->As<ep::CudaStream>()->hip_stream()));
  return *has_not_finite_host;
}

bool HasNotFiniteGpu(ep::Stream* stream, const Blob* blob, bool* has_not_finite_host,
                     bool* has_not_finite_device) {
  const DataType dtype = blob->data_type();
  const int64_t elem_cnt = blob->shape().elem_cnt();
  if (dtype == kFloat) {
    return HasNotFinite<float>(stream, elem_cnt, blob->dptr<float>(), has_not_finite_host,
                               has_not_finite_device);
  } else if (dtype == kDouble) {
    return HasNotFinite<double>(stream, elem_cnt, blob->dptr<double>(), has_not_finite_host,
                                has_not_finite_device);
  } else if (dtype == kFloat16) {
#if __CUDA_ARCH__ >= 530
    return HasNotFinite<half>(stream, elem_cnt, blob->dptr<half>(), has_not_finite_host,
                              has_not_finite_device);
#else
    LOG(FATAL) << "use half need nvcc arch >= 530";
#endif
  } else {
    return false;
  }
}

}  // namespace

CudaCheckNumericsKernelObserver::CudaCheckNumericsKernelObserver()
    : has_not_finite_host_(nullptr), has_not_finite_device_(nullptr) {
  OF_CUDA_CHECK(hipGetDevice(&device_id_));
  OF_CUDA_CHECK(hipHostMalloc(&has_not_finite_host_, sizeof(bool)));
  OF_CUDA_CHECK(hipMalloc(&has_not_finite_device_, sizeof(bool)));
}

CudaCheckNumericsKernelObserver::~CudaCheckNumericsKernelObserver() {
  CudaCurrentDeviceGuard guard(device_id_);
  OF_CUDA_CHECK(hipHostFree(has_not_finite_host_));
  OF_CUDA_CHECK(hipFree(has_not_finite_device_));
}

void CudaCheckNumericsKernelObserver::DidForwardDataContent(KernelContext* ctx,
                                                            const Kernel* kernel) {
  for (const auto& obn : kernel->op_attribute().output_bns()) {
    Blob* blob = ctx->BnInOp2Blob(obn);
    if (blob != nullptr) {
      bool has_not_finite =
          HasNotFiniteGpu(ctx->stream(), blob, has_not_finite_host_, has_not_finite_device_);
      CHECK(!has_not_finite) << kernel->op_conf().name() << " : " << obn << " has nan or inf";
    }
  }
}

}  // namespace oneflow
