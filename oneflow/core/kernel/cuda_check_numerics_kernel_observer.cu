#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/kernel/cuda_check_numerics_kernel_observer.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

template<typename T>
__device__ bool IsNotFinite(T x) {
  return !isfinite(x);
}

template<>
__device__ bool IsNotFinite<half>(half x) {
#if __CUDA_ARCH__ >= 530
  return (__hisinf(x) || __hisnan(x));
#else
  __trap();
  return true;
#endif
}

template<typename T>
__global__ void HasNotFiniteGpuKernel(const int64_t n, const T* x, volatile bool* has_not_finite) {
  if (*has_not_finite) { return; }
  CUDA_1D_KERNEL_LOOP_T(int64_t, i, n) {
    if (IsNotFinite(x[i])) {
      *has_not_finite = true;
      return;
    }
  }
}

template<typename T>
bool HasNotFinite(ep::Stream* stream, const int64_t elem_cnt, const T* data_ptr,
                  bool* has_not_finite_host, bool* has_not_finite_device) {
  OF_CUDA_CHECK(hipMemsetAsync(has_not_finite_device, 0, sizeof(bool),
                                stream->As<ep::CudaStream>()->hip_stream()));
  HasNotFiniteGpuKernel<T>
      <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
         stream->As<ep::CudaStream>()->hip_stream()>>>(elem_cnt, data_ptr, has_not_finite_device);
  OF_CUDA_CHECK(hipMemcpyAsync(has_not_finite_host, has_not_finite_device, sizeof(bool),
                                hipMemcpyDefault, stream->As<ep::CudaStream>()->hip_stream()));
  OF_CUDA_CHECK(hipStreamSynchronize(stream->As<ep::CudaStream>()->hip_stream()));
  return *has_not_finite_host;
}

bool HasNotFiniteGpu(ep::Stream* stream, const Blob* blob, bool* has_not_finite_host,
                     bool* has_not_finite_device) {
  auto* hip_stream = stream->As<ep::CudaStream>();
  const DataType dtype = blob->data_type();
  const int64_t elem_cnt = blob->shape().elem_cnt();
  if (elem_cnt == 0) { return false; }
  if (dtype == kFloat) {
    return HasNotFinite<float>(stream, elem_cnt, blob->dptr<float>(), has_not_finite_host,
                               has_not_finite_device);
  } else if (dtype == kDouble) {
    return HasNotFinite<double>(stream, elem_cnt, blob->dptr<double>(), has_not_finite_host,
                                has_not_finite_device);
  } else if (dtype == kFloat16) {
    if (hip_stream->cuda_arch() >= 530) {
      return HasNotFinite<half>(stream, elem_cnt, blob->dptr<half>(), has_not_finite_host,
                                has_not_finite_device);
    } else {
      LOG(FATAL) << "use half need nvcc arch >= 530";
      return true;
    }
  } else {
    return false;
  }
}

void DumpBlob(KernelContext* ctx, const std::string& bn) {
  Blob* blob = ctx->BnInOp2Blob(bn);
  if (blob != nullptr) {
    std::vector<char> buffer(blob->ByteSizeOfBlobBody());
    OF_CUDA_CHECK(
        hipMemcpy(buffer.data(), blob->dptr(), blob->ByteSizeOfBlobBody(), hipMemcpyDefault));
    OF_CUDA_CHECK(hipDeviceSynchronize());
    std::ofstream ofs(bn);
    ofs.write(buffer.data(), blob->ByteSizeOfBlobBody());
  }
}

void DumpBlobs(KernelContext* ctx, const Kernel* kernel) {
  for (const auto& obn : kernel->op_attribute().output_bns()) { DumpBlob(ctx, obn); }
  for (const auto& ibn : kernel->op_attribute().input_bns()) { DumpBlob(ctx, ibn); }
}

}  // namespace

CudaCheckNumericsKernelObserver::CudaCheckNumericsKernelObserver()
    : has_not_finite_host_(nullptr), has_not_finite_device_(nullptr) {
  OF_CUDA_CHECK(hipGetDevice(&device_id_));
  OF_CUDA_CHECK(hipHostMalloc(&has_not_finite_host_, sizeof(bool)));
  OF_CUDA_CHECK(hipMalloc(&has_not_finite_device_, sizeof(bool)));
}

CudaCheckNumericsKernelObserver::~CudaCheckNumericsKernelObserver() {
  CudaCurrentDeviceGuard guard(device_id_);
  OF_CUDA_CHECK(hipHostFree(has_not_finite_host_));
  OF_CUDA_CHECK(hipFree(has_not_finite_device_));
}

void CudaCheckNumericsKernelObserver::DidForwardDataContent(KernelContext* ctx,
                                                            const Kernel* kernel) {
  for (const auto& obn : kernel->op_attribute().output_bns()) {
    Blob* blob = ctx->BnInOp2Blob(obn);
    if (blob != nullptr) {
      bool has_not_finite =
          HasNotFiniteGpu(ctx->stream(), blob, has_not_finite_host_, has_not_finite_device_);
      if (has_not_finite
          && ParseBooleanFromEnv("ONEFLOW_DEBUG_KERNEL_SYNC_CHECK_NUMERICS_DUMP", false)) {
        DumpBlobs(ctx, kernel);
      }
      CHECK(!has_not_finite) << kernel->op_conf().name() << " : " << obn << " has nan or inf";
    }
  }
}

}  // namespace oneflow
