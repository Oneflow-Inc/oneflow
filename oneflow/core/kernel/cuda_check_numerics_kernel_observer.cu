#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/kernel/cuda_check_numerics_kernel_observer.h"
#include "oneflow/core/kernel/kernel.h"

namespace oneflow {

namespace {

template<typename T>
__device__ bool IsNotFinite(T x) {
  return !isfinite(x);
}

template<>
__device__ bool IsNotFinite<half>(half x) {
  return (__hisinf(x) || __hisnan(x));
}

template<typename T>
__global__ void HasNotFiniteGpuKernel(const int64_t n, const T* x, volatile bool* has_not_finite) {
  if (*has_not_finite) { return; }
  CUDA_1D_KERNEL_LOOP_T(int64_t, i, n) {
    if (IsNotFinite(x[i])) {
      *has_not_finite = true;
      return;
    }
  }
}

template<typename T>
bool HasNotFinite(DeviceCtx* device_ctx, const int64_t elem_cnt, const T* data_ptr,
                  bool* has_not_finite_host, bool* has_not_finite_device) {
  OF_CUDA_CHECK(hipMemsetAsync(has_not_finite_device, 0, sizeof(bool), device_ctx->hip_stream()));
  HasNotFiniteGpuKernel<T>
      <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, device_ctx->hip_stream()>>>(
          elem_cnt, data_ptr, has_not_finite_device);
  OF_CUDA_CHECK(hipMemcpyAsync(has_not_finite_host, has_not_finite_device, sizeof(bool),
                                hipMemcpyDefault, device_ctx->hip_stream()));
  OF_CUDA_CHECK(hipStreamSynchronize(device_ctx->hip_stream()));
  return *has_not_finite_host;
}

bool HasNotFiniteGpu(DeviceCtx* device_ctx, const Blob* blob, bool* has_not_finite_host,
                     bool* has_not_finite_device) {
  const DataType dtype = blob->data_type();
  const int64_t elem_cnt = blob->shape().elem_cnt();
  if (dtype == kFloat) {
    return HasNotFinite<float>(device_ctx, elem_cnt, blob->dptr<float>(), has_not_finite_host,
                               has_not_finite_device);
  } else if (dtype == kDouble) {
    return HasNotFinite<double>(device_ctx, elem_cnt, blob->dptr<double>(), has_not_finite_host,
                                has_not_finite_device);
  } else if (dtype == kFloat16) {
    return HasNotFinite<half>(device_ctx, elem_cnt, blob->dptr<half>(), has_not_finite_host,
                              has_not_finite_device);
  } else {
    return false;
  }
}

}  // namespace

CudaCheckNumericsKernelObserver::CudaCheckNumericsKernelObserver()
    : has_not_finite_host_(nullptr), has_not_finite_device_(nullptr) {
  OF_CUDA_CHECK(hipGetDevice(&device_id_));
  OF_CUDA_CHECK(hipHostMalloc(&has_not_finite_host_, sizeof(bool)));
  OF_CUDA_CHECK(hipMalloc(&has_not_finite_device_, sizeof(bool)));
}

CudaCheckNumericsKernelObserver::~CudaCheckNumericsKernelObserver() {
  CudaCurrentDeviceGuard guard(device_id_);
  OF_CUDA_CHECK(hipHostFree(has_not_finite_host_));
  OF_CUDA_CHECK(hipFree(has_not_finite_device_));
}

void CudaCheckNumericsKernelObserver::DidForwardDataContent(KernelContext* ctx,
                                                            const Kernel* kernel) {
  for (const auto& obn : kernel->op_attribute().output_bns()) {
    Blob* blob = ctx->BnInOp2Blob(obn);
    if (blob != nullptr) {
      bool has_not_finite =
          HasNotFiniteGpu(ctx->device_ctx(), blob, has_not_finite_host_, has_not_finite_device_);
      CHECK(!has_not_finite) << kernel->op_conf().name() << " : " << obn << " has nan or inf";
    }
  }
}

}  // namespace oneflow
