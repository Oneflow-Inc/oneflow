#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/sparse_softmax_cross_entropy_loss_kernel.h"

namespace oneflow {

namespace {

template<typename PredType, typename LabelType>
__global__ void SparseSoftmaxCrossEntropyLossBackwardSub(const int64_t n,
                                                         const int64_t w,
                                                         const LabelType* label,
                                                         PredType* in_diff) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    in_diff[i * w + static_cast<int64_t>(label[i])] -= 1;
  }
}

}  // namespace

template<typename PredType, typename LabelType>
struct SparseSoftmaxCrossEntropyLossKernelUtil<DeviceType::kGPU, PredType,
                                               LabelType> {
  static void BackwardSub(DeviceCtx* ctx, const int64_t n, const int64_t w,
                          const LabelType* label, PredType* in_diff) {
    SparseSoftmaxCrossEntropyLossBackwardSub<PredType>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx->hip_stream()>>>(n, w, label, in_diff);
  }
};

#define MAKE_ENTRY(data_type_pair, label_type_pair)        \
  template struct SparseSoftmaxCrossEntropyLossKernelUtil< \
      DeviceType::kGPU, OF_PP_PAIR_FIRST(data_type_pair),  \
      OF_PP_PAIR_FIRST(label_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_ENTRY, FLOATING_DATA_TYPE_SEQ,
                                 INT_DATA_TYPE_SEQ)
}  // namespace oneflow
