#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/sparse_softmax_cross_entropy_loss_kernel.h"

namespace oneflow {

namespace {

template<typename PredType, typename LabelType>
__global__ void SparseSoftmaxCrossEntropyLossForwardTmp(const int64_t n,
                                                        const int64_t w,
                                                        const LabelType* label,
                                                        const PredType* prob,
                                                        PredType* loss) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    loss[i] = -SAFE_LOG(prob[i * w + static_cast<int64_t>(label[i])]);
  }
}

template<typename PredType, typename LabelType>
__global__ void SparseSoftmaxCrossEntropyLossBackwardSub(const int64_t n,
                                                         const int64_t w,
                                                         const LabelType* label,
                                                         PredType* in_diff) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    in_diff[i * w + static_cast<int64_t>(label[i])] -= 1;
  }
}

}  // namespace

template<typename PredType, typename LabelType>
struct SparseSoftmaxCrossEntropyLossKernelUtil<DeviceType::kGPU, PredType,
                                               LabelType> {
  static void ComputeLoss(DeviceCtx* ctx, const int64_t n, const int64_t w,
                          const LabelType* label, const PredType* prob,
                          PredType* loss) {
    SparseSoftmaxCrossEntropyLossForwardTmp<PredType>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx->hip_stream()>>>(n, w, label, prob, loss);
  }

  static void BackwardSub(DeviceCtx* ctx, const int64_t n, const int64_t w,
                          const LabelType* label, PredType* in_diff) {
    SparseSoftmaxCrossEntropyLossBackwardSub<PredType>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx->hip_stream()>>>(n, w, label, in_diff);
  }
};

#define MAKE_ENTRY(data_type_pair, label_type_pair)        \
  template struct SparseSoftmaxCrossEntropyLossKernelUtil< \
      DeviceType::kGPU, OF_PP_PAIR_FIRST(data_type_pair),  \
      OF_PP_PAIR_FIRST(label_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_ENTRY, FLOATING_DATA_TYPE_SEQ,
                                 INT_DATA_TYPE_SEQ)
}  // namespace oneflow
