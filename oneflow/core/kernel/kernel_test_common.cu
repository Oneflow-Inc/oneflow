#include <random>
#include "oneflow/core/device/cuda_device_context.h"
#include "oneflow/core/kernel/kernel_test_common.h"

namespace oneflow {

namespace test {

template<typename FloatingPointType>
class KernelTestCommon<DeviceType::kGPU, FloatingPointType> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(KernelTestCommon);
  KernelTestCommon() = delete;

  static Blob* CreateBlobWithVector(const std::vector<int64_t>& dim_vec,
                                    FloatingPointType* data_vec) {
    void* dptr;
    Shape* shape = new Shape(dim_vec);
    size_t dptr_size = shape->elem_cnt() * sizeof(FloatingPointType);
    CudaCheck(hipMalloc(&dptr, dptr_size));
    CudaCheck(hipMemcpy(dptr, data_vec, dptr_size, hipMemcpyHostToDevice));
    return new Blob(dptr, shape);
  }

  static Blob* CreateBlobWithSameValue(const std::vector<int64_t>& dim_vec,
                                       FloatingPointType value) {
    Shape* shape = new Shape(dim_vec);
    FloatingPointType* data_vec = new FloatingPointType[shape->elem_cnt()];
    std::fill(data_vec, data_vec + shape->elem_cnt(), value);
    return CreateBlobWithVector(dim_vec, data_vec);
  }

  static Blob* CreateBlobWithRandomValue(const std::vector<int64_t>& dim_vec) {
    Shape* shape = new Shape(dim_vec);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<FloatingPointType> dis(0, 10);
    FloatingPointType* data_vec = new FloatingPointType[shape->elem_cnt()];
    for (int64_t i = 0; i != shape->elem_cnt(); ++i) { data_vec[i] = dis(gen); }
    return CreateBlobWithVector(dim_vec, data_vec);
  }

  static void BuildKernelCtx(KernelCtx* ctx) {
    hipStream_t* hip_stream = new hipStream_t;
    hipblasHandle_t* cublas_handle = new hipblasHandle_t;
    CudaCheck(hipStreamCreate(hip_stream));
    CudaCheck(hipblasCreate(cublas_handle));
    CudaCheck(hipblasSetStream(*cublas_handle, *hip_stream));
    ctx->device_ctx = new CudaDeviceCtx(hip_stream, cublas_handle, nullptr);
  }

  static void SyncStream(KernelCtx* ctx) {
    CudaCheck(hipStreamSynchronize(ctx->device_ctx->hip_stream()));
  }

  static void BlobCmp(Blob* lhs, Blob* rhs) {
    using KTCommonCpu = KernelTestCommon<DeviceType::kCPU, FloatingPointType>;

    FloatingPointType* dptr;
    size_t dptr_size = lhs->shape().elem_cnt() * sizeof(FloatingPointType);
    CudaCheck(hipHostMalloc(&dptr, dptr_size));
    memset(dptr, 0, dptr_size);
    Blob* copy_lhs =
        KTCommonCpu::CreateBlobWithVector(lhs->shape().dim_vec(), dptr);
    Blob* copy_rhs =
        KTCommonCpu::CreateBlobWithVector(rhs->shape().dim_vec(), dptr);
    CudaCheck(hipMemcpy(copy_lhs->mut_dptr(), lhs->dptr(), dptr_size,
                         hipMemcpyDeviceToHost));
    CudaCheck(hipMemcpy(copy_rhs->mut_dptr(), rhs->dptr(), dptr_size,
                         hipMemcpyDeviceToHost));

    KTCommonCpu::BlobCmp(copy_lhs, copy_rhs);
  }

  static void CheckResult(
      std::function<Blob*(const std::string&)> BnInOp2BlobPtr,
      const std::string& check, const std::string& expected) {
    BlobCmp(BnInOp2BlobPtr(check), BnInOp2BlobPtr(expected));
  }
};

template class KernelTestCommon<DeviceType::kGPU, float>;
template class KernelTestCommon<DeviceType::kGPU, double>;

}  // namespace test
}  // namespace oneflow
