#include <random>
#include "oneflow/core/device/cuda_device_context.h"
#include "oneflow/core/kernel/kernel_test_common.h"

namespace oneflow {

namespace test {

template<typename FloatingPointType>
class KernelTestCommon<DeviceType::kGPU, FloatingPointType> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(KernelTestCommon);
  KernelTestCommon() = delete;

  static Blob* CreateBlobWithVector(const std::vector<int64_t>& dim_vec,
                                    FloatingPointType* data_vec) {
    void* dptr;
    Shape* shape = new Shape(dim_vec);
    size_t dptr_size = shape->elem_cnt() * sizeof(FloatingPointType);
    CudaCheck(hipMalloc(&dptr, dptr_size));
    CudaCheck(hipMemcpy(dptr, data_vec, dptr_size, hipMemcpyHostToDevice));
    return new Blob(dptr, shape);
  }

  static Blob* CreateBlobWithSameValue(const std::vector<int64_t>& dim_vec,
                                       FloatingPointType value) {
    Shape* shape = new Shape(dim_vec);
    FloatingPointType* data_vec = new FloatingPointType[shape->elem_cnt()];
    std::fill(data_vec, data_vec + shape->elem_cnt(), value);
    return CreateBlobWithVector(dim_vec, data_vec);
  }

  static Blob* CreateBlobWithRandomValue(const std::vector<int64_t>& dim_vec) {
    Shape* shape = new Shape(dim_vec);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<FloatingPointType> dis(0, 10);
    FloatingPointType* data_vec = new FloatingPointType[shape->elem_cnt()];
    for (int64_t i = 0; i != shape->elem_cnt(); ++i) { data_vec[i] = dis(gen); }
    return CreateBlobWithVector(dim_vec, data_vec);
  }

  static void BuildKernelCtx(KernelCtx* ctx) {
    hipStream_t* hip_stream = new hipStream_t;
    hipblasHandle_t* cublas_handle = new hipblasHandle_t;
    CudaCheck(hipStreamCreate(hip_stream));
    CudaCheck(hipblasCreate(cublas_handle));
    CudaCheck(hipblasSetStream(*cublas_handle, *hip_stream));
    ctx->device_ctx = new CudaDeviceCtx(hip_stream, cublas_handle, nullptr);
  }

  static void SyncStream(KernelCtx* ctx) {
    CudaCheck(hipStreamSynchronize(ctx->device_ctx->hip_stream()));
  }

  static void BlobCmp(Blob* lhs, Blob* rhs) {
    using KTCommonCpu = KernelTestCommon<DeviceType::kCPU, FloatingPointType>;

    FloatingPointType* dptr;
    size_t dptr_size = lhs->shape().elem_cnt() * sizeof(FloatingPointType);
    CudaCheck(hipHostMalloc(&dptr, dptr_size));
    memset(dptr, 0, dptr_size);
    Blob* copy_lhs =
        KTCommonCpu::CreateBlobWithVector(lhs->shape().dim_vec(), dptr);
    Blob* copy_rhs =
        KTCommonCpu::CreateBlobWithVector(rhs->shape().dim_vec(), dptr);
    CudaCheck(hipMemcpy(copy_lhs->mut_dptr(), lhs->dptr(), dptr_size,
                         hipMemcpyDeviceToHost));
    CudaCheck(hipMemcpy(copy_rhs->mut_dptr(), rhs->dptr(), dptr_size,
                         hipMemcpyDeviceToHost));

    KTCommonCpu::BlobCmp(copy_lhs, copy_rhs);
  }

  static void CheckResult(
      std::function<Blob*(const std::string&)> BnInOp2BlobPtr,
      const std::string& check, const std::string& expected) {
    BlobCmp(BnInOp2BlobPtr(check), BnInOp2BlobPtr(expected));
  }

  static void CheckFillResult(const Blob& check_blob,
                              const FillConf& fill_conf) {
    using KTCommonCpu = KernelTestCommon<DeviceType::kCPU, FloatingPointType>;

    void* host_raw_dptr = nullptr;
    size_t byte_size =
        check_blob.shape().elem_cnt() * sizeof(FloatingPointType);
    CudaCheck(hipHostMalloc(&host_raw_dptr, byte_size));
    memset(host_raw_dptr, 0, byte_size);
    std::unique_ptr<void, std::function<void(void*)>> host_unique_ptr(
        host_raw_dptr, [&](void* dptr) { CudaCheck(hipHostFree(dptr)); });
    std::unique_ptr<Shape> host_blob_shape(new Shape(check_blob.shape()));

    std::unique_ptr<Blob> copy_check_blob(
        new Blob(host_unique_ptr.get(), host_blob_shape.get()));
    CudaCheck(hipMemcpy(copy_check_blob->mut_dptr(), check_blob.dptr(),
                         byte_size, hipMemcpyDeviceToHost));

    KTCommonCpu::CheckFillResult(*copy_check_blob, fill_conf);
  }
};  // namespace test

template class KernelTestCommon<DeviceType::kGPU, float>;
template class KernelTestCommon<DeviceType::kGPU, double>;

}  // namespace test
}  // namespace oneflow
