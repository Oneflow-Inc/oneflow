#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/salr_model_update_kernel.h"

namespace oneflow {

namespace {

// if diff(t) * diff(t-1) > 0
// then learning_rate = learning_rate + delta
// else learning_rate = learning_rate * (1 - delta)
template<typename FloatingPointType>
__global__ void UpdateLearningRateGpu(const int64_t n,
                                      const FloatingPointType delta,
                                      FloatingPointType* last_diff_flag,
                                      const FloatingPointType* model_diff,
                                      FloatingPointType* learning_rate) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    if (last_diff_flag[i] * model_diff[i] > 0) {
      learning_rate[i] = learning_rate[i] + delta;
    } else {
      learning_rate[i] = learning_rate[i] * (1 - delta);
    }
    last_diff_flag[i] = model_diff[i] > 0 ? 1 : -1;
  }
}

// model -= (-epsilon) * learning_rate * model_diff
template<typename FloatingPointType>
__global__ void UpdateModelGpu(const int64_t n, FloatingPointType* model,
                               const FloatingPointType* model_diff,
                               const FloatingPointType* learning_rate,
                               const FloatingPointType epsilon) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    model[i] -= -epsilon * learning_rate[i] * model_diff[i];
  }
}

}  // namespace

template<typename FloatingPointType>
class SALRMdUpdateKernelUtil<DeviceType::kGPU, FloatingPointType> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(SALRMdUpdateKernelUtil);
  SALRMdUpdateKernelUtil() = delete;

  static void UpdateLearningRate(const KernelCtx& ctx, const int64_t n,
                                 const FloatingPointType delta,
                                 FloatingPointType* last_diff_flag,
                                 const FloatingPointType* model_diff,
                                 FloatingPointType* learning_rate) {
    UpdateLearningRateGpu<FloatingPointType>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx.device_ctx->hip_stream()>>>(n, delta, last_diff_flag,
                                            model_diff, learning_rate);
  }

  static void UpdateModel(const KernelCtx& ctx, const int64_t n,
                          FloatingPointType* model,
                          const FloatingPointType* model_diff,
                          const FloatingPointType* learning_rate,
                          const FloatingPointType epsilon) {
    UpdateModelGpu<FloatingPointType>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx.device_ctx->hip_stream()>>>(n, model, model_diff, learning_rate,
                                            epsilon);
  }
};

INSTANTIATE_GPU_KERNEL_UTIL_CLASS(SALRMdUpdateKernelUtil);

}  // namespace oneflow
