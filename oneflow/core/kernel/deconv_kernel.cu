#include "oneflow/core/operator/deconv_op.h"
#include "oneflow/core/kernel/deconv_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/device/cudnn_conv_util.h"

namespace oneflow {

template<typename T>
class DeconvGPUKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(DeconvGPUKernel);
  DeconvGPUKernel() = default;
  ~DeconvGPUKernel() = default;

 private:
  const PbMessage& GetCustomizedOpConf() const override { return this->op_conf().deconv_conf(); }

  void VirtualKernelInit() override {
    const DeconvOpConf& op_conf = this->op_conf().deconv_conf();
    const ConvConf& conv_conf = this->op_conf().deconv_conf().conv_conf();
    const int32_t num_spatial_dims = this->op_conf().deconv_conf().conv_conf().num_spatial_dims();
    DenseShapeView x_shape(this->kernel_conf().deconv_conf().in());
    DenseShapeView y_shape(this->kernel_conf().deconv_conf().out());
    DenseShapeView weight_shape(this->kernel_conf().deconv_conf().weight());

    const std::string& data_format = conv_conf.data_format();
    this->x_desc_.reset(new CudnnTensorDesc(GetDataType<T>::value, x_shape, data_format));
    this->y_desc_.reset(new CudnnTensorDesc(GetDataType<T>::value, y_shape, data_format));
    this->filter_desc_.reset(new CudnnFilterDesc(GetDataType<T>::value, weight_shape, data_format));
    this->deconv_desc_.reset(new CudnnDeconvDesc(GetDataType<T>::value, x_shape,
                                                 this->op_conf().deconv_conf().conv_conf()));
    if (op_conf.use_bias()) {
      int32_t filters = op_conf.filters();
      if (num_spatial_dims == 2) {
        if (data_format == "channels_first") {
          this->bias_desc_.reset(
              new CudnnTensorDesc(HIPDNN_TENSOR_NCHW, GetDataType<T>::value, 1, filters, 1, 1));
        } else if (data_format == "channels_last") {
          if (GetDataType<T>::value == DataType::kDouble) {
            LOG(FATAL) << "CUDNN 1d & 2d support channels last only if data type "
                          "is float";
          }
          this->bias_desc_.reset(
              new CudnnTensorDesc(HIPDNN_TENSOR_NHWC, GetDataType<T>::value, 1, filters, 1, 1));
        } else {
          UNIMPLEMENTED();
        }
      } else {
        if (data_format == "channels_last") {
          LOG(FATAL) << "CUDNN Nd API only support channels first";
        }
        std::vector<int32_t> bias_dim(num_spatial_dims + 2, 1);
        std::vector<int32_t> stride_of_bias_tensor(num_spatial_dims + 2, 1);
        bias_dim[1] = filters;
        stride_of_bias_tensor[0] = filters;
        this->bias_desc_.reset(new CudnnTensorDesc(GetDataType<T>::value, num_spatial_dims + 2,
                                                   bias_dim.data(), stride_of_bias_tensor.data()));
      }
    }
  }

  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    CudnnConvArgs args(this->op_conf().deconv_conf().conv_conf(), ctx.device_ctx->cudnn_handle(),
                       BnInOp2Blob("y"), BnInOp2Blob("x"), BnInOp2Blob("filter"),
                       BnInOp2Blob("cudnn_buf"),
                       this->job_desc().job_conf().cudnn_conv_use_deterministic_algo_only(),
                       this->job_desc().job_conf().cudnn_conv_heuristic_search_algo());
    hipdnnConvolutionBwdDataAlgo_t algo;
    size_t work_space_size = 0;
    if (this->job_desc().job_conf().has_cudnn_conv_force_bwd_data_algo()) {
      algo = static_cast<hipdnnConvolutionBwdDataAlgo_t>(
          this->job_desc().job_conf().cudnn_conv_force_bwd_data_algo());
      CudaCheck(GetConvWorkspaceSize(args, algo, &work_space_size));
    } else {
      auto algo_perf = FindCudnnConvAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>(args);
      algo = algo_perf->algo;
      work_space_size = algo_perf->memory;
    }
    CHECK_LE(work_space_size, BnInOp2Blob("cudnn_buf")->ByteSizeOfBlobBody());
    CudaCheck(hipdnnConvolutionBackwardData(args.handle, CudnnSPOnePtr<T>(), args.wdesc.Get(),
                                           args.w_dptr, args.ydesc.Get(), args.y_dptr,
                                           args.cdesc.Get(), algo, args.work_space, work_space_size,
                                           CudnnSPZeroPtr<T>(), args.xdesc.Get(), args.x_dptr));
  }

  mutable std::unique_ptr<CudnnTensorDesc> x_desc_;
  mutable std::unique_ptr<CudnnTensorDesc> y_desc_;
  mutable std::unique_ptr<CudnnFilterDesc> filter_desc_;
  mutable std::unique_ptr<CudnnDeconvDesc> deconv_desc_;
  mutable std::unique_ptr<CudnnTensorDesc> bias_desc_;
};

#define REGISTER_DECONV_GPU_KERNEL(dtype)                                                   \
  REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kDeconvConf, DeviceType::kGPU, dtype, \
                                        DeconvGPUKernel<dtype>)

REGISTER_DECONV_GPU_KERNEL(float);
REGISTER_DECONV_GPU_KERNEL(double);
REGISTER_DECONV_GPU_KERNEL(float16);

}  //  namespace oneflow
