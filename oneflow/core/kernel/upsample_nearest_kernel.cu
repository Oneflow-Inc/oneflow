#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/upsample_nearest_kernel.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void UpsampleNearestForward(const int64_t nthreads, const T* in_dptr,
                                       const int64_t channel_num, const int64_t height,
                                       const int64_t width, const int64_t new_height,
                                       const int64_t new_width, const float scale_h,
                                       const float scale_w, const bool align_corners, T* out_dptr) {
  const int64_t new_area = new_height * new_width;
  const int64_t channel_area = channel_num * height * width;
  const int64_t channel_new_area = channel_num * new_height * new_width;
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t h = (index / new_width) % new_height;
    const int64_t w = index % new_width;
    const int64_t c = (index / new_area) % channel_num;
    const int64_t n = index / channel_new_area;
    const int64_t in_h = min((align_corners) ? static_cast<int64_t>(roundf(h * scale_h))
                                             : static_cast<int64_t>(floorf(h * scale_h)),
                             height - 1);
    const int64_t in_w = min((align_corners) ? static_cast<int64_t>(roundf(w * scale_w))
                                             : static_cast<int64_t>(floorf(w * scale_w)),
                             width - 1);
    out_dptr[index] = in_dptr[n * channel_area + (c * height + in_h) * width + in_w];
  }
}

template<typename T>
__global__ void UpsampleNearestBackward(const int64_t nthreads, const T* dy_dptr,
                                        const int64_t channel_num, const int64_t height,
                                        const int64_t width, const int64_t new_height,
                                        const int64_t new_width, const float scale_h,
                                        const float scale_w, const bool align_corners, T* dx_dptr) {
  const int64_t area = height * width;
  const int64_t new_area = new_height * new_width;
  const int64_t channel_area = channel_num * height * width;
  const int64_t channel_new_area = channel_num * new_height * new_width;
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t h = (index / new_width) % new_height;
    const int64_t w = index % new_width;
    const int64_t c = (index / new_area) % channel_num;
    const int64_t n = index / channel_new_area;
    const int64_t in_h = min((align_corners) ? static_cast<int64_t>(roundf(h * scale_h))
                                             : static_cast<int64_t>(floorf(h * scale_h)),
                             height - 1);
    const int64_t in_w = min((align_corners) ? static_cast<int64_t>(roundf(w * scale_w))
                                             : static_cast<int64_t>(floorf(w * scale_w)),
                             width - 1);
    gpu_atomic_add(dx_dptr + n * channel_area + (c * height + in_h) * width + in_w, dy_dptr[index]);
  }
}

}  // namespace

template<typename T>
struct UpsampleNearestUtil<DeviceType::kGPU, T> {
  static void Forward(const KernelCtx& ctx, const const float scale_h, const float scale_w,
                      const bool align_corners, const Blob* in_blob, Blob* out_blob) {
    const int64_t elem_cnt = out_blob->shape().elem_cnt();
    UpsampleNearestForward<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                ctx.device_ctx->hip_stream()>>>(
        elem_cnt, in_blob->dptr<T>(), in_blob->shape().At(1), in_blob->shape().At(2),
        in_blob->shape().At(3), out_blob->shape().At(2), out_blob->shape().At(3), scale_h, scale_w,
        align_corners, out_blob->mut_dptr<T>());
  }

  static void Backward(const KernelCtx& ctx, const const float scale_h, const float scale_w,
                       const bool align_corners, const Blob* dy_blob, Blob* dx_blob) {
    const int64_t elem_cnt = dy_blob->shape().elem_cnt();
    UpsampleNearestBackward<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                 ctx.device_ctx->hip_stream()>>>(
        elem_cnt, dy_blob->dptr<T>(), dx_blob->shape().At(1), dx_blob->shape().At(2),
        dx_blob->shape().At(3), dy_blob->shape().At(2), dy_blob->shape().At(3), scale_h, scale_w,
        align_corners, dx_blob->mut_dptr<T>());
  }
};

#define INSTANTIATE_UPSAMPLE_NEAREST_KERNEL_UTIL(type_cpp, type_proto) \
  template class UpsampleNearestUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_UPSAMPLE_NEAREST_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ);

}  // namespace oneflow
