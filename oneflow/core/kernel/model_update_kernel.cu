#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/model_update_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void L1RegularizationGpu(const int64_t n, float a, const T* x,
                                    T* y) {
  T zero = static_cast<T>(0);
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] += a * static_cast<T>((x[i] > zero) - (zero < x[i]));
  }
}

}  // namespace

template<typename T>
class MdUpdateKernelUtil<DeviceType::kGPU, T> final {
 public:
  static void L1Regularization(DeviceCtx* ctx, int64_t n, float weight_decay,
                               const T* model, T* model_diff) {
    L1RegularizationGpu<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx->hip_stream()>>>(n, weight_decay, model, model_diff);
  }
};

#define INSTANTIATE_GPU_KERNEL_UTIL(type_cpp, type_proto) \
  template class MdUpdateKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GPU_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
