#include "oneflow/core/kernel/boxing_kernel.h"
#include <string>
#include "oneflow/core/operator/op_conf.pb.h"

namespace oneflow {

template<typename floating_point_type>
void BoxingKernel<DeviceType::kGPU, floating_point_type>::OFMemcpy(
    const KernelCtx& ctx, void* dst, const void* src, size_t sz) {
  CHECK_EQ(hipMemcpyAsync(dst, src, sz, hipMemcpyDeviceToDevice, 
      ctx.device_ctx->hip_stream()), hipSuccess);
}

template<typename floating_point_type>
void BoxingKernel<DeviceType::kGPU, floating_point_type>::OFBlobCpy(
    const KernelCtx& ctx, const Blob* a, Blob* b) {
  CHECK_EQ(hipMemcpyAsync(static_cast<floating_point_type*>(b->mut_dptr()), \
        static_cast<const floating_point_type*>(a->dptr()), \
        sizeof(floating_point_type) * a->shape().elem_cnt(), \
        hipMemcpyDeviceToDevice, ctx.device_ctx->hip_stream()), \
      hipSuccess);
}

template<>
void BoxingKernel<DeviceType::kGPU, float>::OFBlobAdd(
    const KernelCtx& ctx, const Blob* a, Blob* b) {
  static const float alpha = 1.0;
  CHECK_EQ(hipblasSaxpy(
        ctx.device_ctx->cublas_handle(),
        a->shape().elem_cnt(), &alpha,
        static_cast<const float*>(a->dptr()), 1,
        static_cast<float*>(b->mut_dptr()), 1), \
      hipSuccess);
}

template<>
void BoxingKernel<DeviceType::kGPU, double>::OFBlobAdd(
    const KernelCtx& ctx, const Blob* a, Blob* b) {
  static const double alpha = 1.0;
  CHECK_EQ(hipblasDaxpy(
        ctx.device_ctx->cublas_handle(),
        a->shape().elem_cnt(), &alpha,
        static_cast<const double*>(a->dptr()), 1,
        static_cast<double*>(b->mut_dptr()), 1), 
      hipSuccess);
}

template<>
void BoxingKernel<DeviceType::kGPU, float>::OFBlasAxpy(
    const KernelCtx& ctx, const int N, const float alpha, const float *X, \
    const int incX, float *Y, const int incY) {
  float tmp_alpha = alpha;
  CHECK_EQ(hipblasSaxpy(
        ctx.device_ctx->cublas_handle(),
        N, &tmp_alpha, X, incX, Y, incY), 
      hipSuccess);
}

template<>
void BoxingKernel<DeviceType::kGPU, double>::OFBlasAxpy(
    const KernelCtx& ctx, const int N, const double alpha, const double *X, \
    const int incX, double *Y, const int incY) {
  double tmp_alpha = alpha;
  CHECK_EQ(hipblasDaxpy(
        ctx.device_ctx->cublas_handle(),
        N, &tmp_alpha, X, incX, Y, incY), 
      hipSuccess);
}

template<> 
void BoxingKernel<DeviceType::kGPU, float>::OFBlasScal(
    const KernelCtx& ctx, const int n, const float alpha, float* x, int incx) {
  float tmp_alpha = alpha;
  CHECK_EQ(hipblasSscal(
        ctx.device_ctx->cublas_handle(),
        n, &tmp_alpha, x, incx), 
      hipSuccess);
}

template<> 
void BoxingKernel<DeviceType::kGPU, double>::OFBlasScal(
    const KernelCtx& ctx, const int n, const double alpha, double* x, \
    int incx) {
  double tmp_alpha = alpha;
  CHECK_EQ(hipblasDscal(
        ctx.device_ctx->cublas_handle(),
        n, &tmp_alpha, x, incx), 
      hipSuccess);
}

INSTANTIATE_GPU_KERNEL_CLASS(BoxingKernel);
REGISTER_GPU_KERNEL(OperatorConf::kBoxingConf, BoxingKernel);
}  // namespace oneflow
