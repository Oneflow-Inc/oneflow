#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/rmsprop_model_update_kernel.h"

namespace oneflow {

namespace {

template<typename T, bool centered>
__global__ void UpdateModelGpu(int64_t n, const int64_t* train_step, const float* learning_rate,
                               T decay_rate, T epsilon, T weight_decay, const T* model_diff,
                               T* model, T* mean_square, T* mean_gradient) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    T model_diff_val = model_diff[i];
    T mean_square_val = mean_square[i];
    mean_square_val =
        (1 - decay_rate) * model_diff_val * model_diff_val + decay_rate * mean_square_val;
    mean_square[i] = mean_square_val;
    T denom_t;
    if (centered) {
      T mean_gradient_val = mean_gradient[i];
      mean_gradient_val = (1 - decay_rate) * model_diff_val + decay_rate * mean_gradient_val;
      mean_gradient[i] = mean_gradient_val;
      denom_t = mean_square_val - mean_gradient_val * mean_gradient_val;
    } else {
      denom_t = mean_square_val;
    }
    model[i] = model[i] - *learning_rate * model_diff_val * rsqrt(denom_t + epsilon);
  }
}

}  // namespace

template<typename T>
class RMSPropMdUpdateKernelUtil<DeviceType::kGPU, T> final {
 public:
  static void UpdateModel(DeviceCtx* ctx, int64_t n, const int64_t* train_step,
                          const float* learning_rate, T decay_rate, T epsilon, bool centered,
                          T weight_decay, const T* model_diff, T* model, T* mean_square,
                          T* mean_gradient) {
    if (centered) {
      UpdateModelGpu<T, true>
          <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              n, train_step, learning_rate, decay_rate, epsilon, weight_decay, model_diff, model,
              mean_square, mean_gradient);
    } else {
      UpdateModelGpu<T, false>
          <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              n, train_step, learning_rate, decay_rate, epsilon, weight_decay, model_diff, model,
              mean_square, mean_gradient);
    }
  }
};

#define INSTANTIATE_GPU_KERNEL_UTIL(type_cpp, type_proto) \
  template class RMSPropMdUpdateKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GPU_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
