#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/rmsprop_model_update_kernel.h"

namespace oneflow {

namespace {

template<typename FloatingPointType>
__global__ void UpdateMeanSquareGpu(const int64_t n,
                                    const FloatingPointType alpha,
                                    const FloatingPointType decay_rate,
                                    FloatingPointType* mean_square,
                                    const FloatingPointType* model_diff) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    mean_square[i] =
        alpha * model_diff[i] * model_diff[i] + decay_rate * mean_square[i];
  }
}

template<typename FloatingPointType>
__global__ void UpdateModelGpu(const int64_t n, FloatingPointType* model,
                               const FloatingPointType* model_diff,
                               const FloatingPointType* mean_square,
                               const FloatingPointType epsilon,
                               const FloatingPointType alpha) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    model[i] -= alpha * model_diff[i] / std::sqrt(mean_square[i] + epsilon);
  }
}

}  // namespace

template<typename FloatingPointType>
class RMSPropMdUpdateKernelUtil<DeviceType::kGPU, FloatingPointType> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(RMSPropMdUpdateKernelUtil);
  RMSPropMdUpdateKernelUtil() = delete;

  static void UpdateMeanSquare(const KernelCtx& ctx, const int64_t n,
                               const FloatingPointType alpha,
                               const FloatingPointType decay_rate,
                               FloatingPointType* mean_square,
                               const FloatingPointType* model_diff) {
    UpdateMeanSquareGpu<FloatingPointType>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx.device_ctx->hip_stream()>>>(n, alpha, decay_rate, mean_square,
                                            model_diff);
  }

  static void UpdateModel(const KernelCtx& ctx, const int64_t n,
                          FloatingPointType* model,
                          const FloatingPointType* model_diff,
                          const FloatingPointType* mean_square,
                          const FloatingPointType epsilon,
                          const FloatingPointType alpha) {
    UpdateModelGpu<FloatingPointType>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx.device_ctx->hip_stream()>>>(n, model, model_diff, mean_square,
                                            epsilon, alpha);
  }
};

INSTANTIATE_GPU_KERNEL_UTIL_CLASS(RMSPropMdUpdateKernelUtil);

}  // namespace oneflow
