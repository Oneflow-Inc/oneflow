#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/rmsprop_model_update_kernel.h"

namespace oneflow {

namespace {

// mean_square = decay_rate * mean_square + (1 - decay_rate) * model_diff ^ 2
template<typename FloatingPointType>
__global__ void UpdateMeanSquareGpu(const int64_t n,
                                    const FloatingPointType decay_rate,
                                    FloatingPointType* mean_square,
                                    const FloatingPointType* model_diff) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    mean_square[i] = decay_rate * mean_square[i]
                     + (1 - decay_rate) * model_diff[i] * model_diff[i];
  }
}

// model -= alpha * model_diff / (sqrt(mean_square) + epsilon)
template<typename FloatingPointType>
__global__ void UpdateModelGpu(const int64_t n, FloatingPointType* model,
                               const FloatingPointType* model_diff,
                               const FloatingPointType* mean_square,
                               const FloatingPointType epsilon,
                               const FloatingPointType alpha) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    model[i] -= alpha * model_diff[i] / (std::sqrt(mean_square[i]) + epsilon);
  }
}

}  // namespace

template<typename FloatingPointType>
class RMSPropMdUpdateKernelUtil<DeviceType::kGPU, FloatingPointType> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(RMSPropMdUpdateKernelUtil);
  RMSPropMdUpdateKernelUtil() = delete;

  static void UpdateMeanSquare(const KernelCtx& ctx, const int64_t n,
                               const FloatingPointType decay_rate,
                               FloatingPointType* mean_square,
                               const FloatingPointType* model_diff) {
    UpdateMeanSquareGpu<FloatingPointType>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx.device_ctx->hip_stream()>>>(n, decay_rate, mean_square,
                                            model_diff);
  }

  static void UpdateModel(const KernelCtx& ctx, const int64_t n,
                          FloatingPointType* model,
                          const FloatingPointType* model_diff,
                          const FloatingPointType* mean_square,
                          const FloatingPointType epsilon,
                          const FloatingPointType alpha) {
    UpdateModelGpu<FloatingPointType>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx.device_ctx->hip_stream()>>>(n, model, model_diff, mean_square,
                                            epsilon, alpha);
  }
};

INSTANTIATE_GPU_KERNEL_UTIL_CLASS(RMSPropMdUpdateKernelUtil);

}  // namespace oneflow
