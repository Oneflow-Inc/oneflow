#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/roi_align_kernel.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T>
__device__ T BilinearInterpolate(const T* channel_dptr, const int32_t height, const int32_t width,
                                 const T y, const T x) {
  if (y < -1.0 || y > height || x < -1.0 || x > width) { return 0; }

  const int32_t y_low = (y <= 0) ? 0 : y;
  const int32_t x_low = (x <= 0) ? 0 : x;

  if (y_low >= height - 1 || x_low >= width - 1) { return 0; }
  const int32_t y_high = y_low + 1;
  const int32_t x_high = x_low + 1;

  const T ly = y - y_low;
  const T lx = x - x_low;
  const T hy = y_high - y;
  const T hx = x_high - x;

  // https://en.wikipedia.org/wiki/Bilinear_interpolation
  const int64_t q11 = y_low * width + x_low;
  const int64_t q21 = y_low * width + x_high;
  const int64_t q12 = y_high * width + x_low;
  const int64_t q22 = y_high * width + x_high;
  //  no 1 / (x_high - x_low) * (y_high - y_low) because it will always be 1 in RoI Align
  return (hy * hx) * channel_dptr[q11] + (hy * lx) * channel_dptr[q21]
         + (ly * hx) * channel_dptr[q12] + (ly * lx) * channel_dptr[q22];
}

template<typename T>
__device__ bool BilinearInterpolateDiff(const T bin_diff_avg, const int64_t height,
                                        const int64_t width, const T y, const T x, T& diff11,
                                        T& diff21, T& diff12, T& diff22, int32_t& x_low,
                                        int32_t& x_high, int32_t& y_low, int32_t& y_high) {
  if (y < -1.0 || y > height || x < -1.0 || x > width) { return false; }

  if (y > 0) { y_low = y; }
  if (x > 0) { x_low = x; }

  if (y_low >= height - 1 || x_low >= width - 1) { return false; }
  y_high = y_low + 1;
  x_high = x_low + 1;

  const T ly = y - y_low;
  const T lx = x - x_low;
  const T hy = y_high - y;
  const T hx = x_high - x;

  diff11 = bin_diff_avg * hy * hx;
  diff21 = bin_diff_avg * hy * lx;
  diff12 = bin_diff_avg * ly * hx;
  diff22 = bin_diff_avg * ly * lx;
  return true;
}

template<typename T>
__global__ void RoIAlignForward(const int64_t nthreads, const T* in_dptr, const float spatial_scale,
                                const int32_t sampling_ratio, const int64_t channel_num,
                                const int64_t height, const int64_t width, const int64_t roi_num,
                                const int64_t pooled_height, const int64_t pooled_width,
                                const T* rois_dptr, T* out_dptr) {
  const int64_t pooled_area = pooled_height * pooled_width;
  const int64_t channel_pooled_area = channel_num * pooled_height * pooled_width;
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t h = (index / pooled_width) % pooled_height;
    const int64_t w = index % pooled_width;
    const int64_t c = (index / pooled_area) % channel_num;
    const int64_t r = (index / channel_pooled_area) % roi_num;
    const int64_t n = index / channel_pooled_area / roi_num;
    const T* offset_rois_dptr = rois_dptr + (n * roi_num + r) * 4;
    const T roi_start_w = offset_rois_dptr[0] * spatial_scale;
    const T roi_start_h = offset_rois_dptr[1] * spatial_scale;
    const T roi_end_w = offset_rois_dptr[2] * spatial_scale;
    const T roi_end_h = offset_rois_dptr[3] * spatial_scale;
    const T roi_height = max(roi_end_h - roi_start_h, static_cast<T>(1.0));
    const T roi_width = max(roi_end_w - roi_start_w, static_cast<T>(1.0));
    const T bin_height = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    const T bin_width = static_cast<T>(roi_width) / static_cast<T>(pooled_width);
    const int32_t bin_grid_height =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height);
    const int32_t bin_grid_width =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);
    const T bin_grid_density_h = bin_height / static_cast<T>(bin_grid_height);
    const T bin_grid_density_w = bin_width / static_cast<T>(bin_grid_width);

    const T* channel_dptr = in_dptr + (n * channel_num + c) * height * width;
    T out_val = 0.0;
    FOR_RANGE(int64_t, grid_i, 0, bin_grid_height) {
      // + .5f for center position
      const T y = roi_start_h + h * bin_height + static_cast<T>(grid_i + 0.5f) * bin_grid_density_h;
      FOR_RANGE(int64_t, grid_j, 0, bin_grid_width) {
        const T x =
            roi_start_w + w * bin_width + static_cast<T>(grid_j + 0.5f) * bin_grid_density_w;
        out_val += BilinearInterpolate(channel_dptr, height, width, y, x);
      }
    }
    // average pooling
    out_dptr[index] = out_val / (bin_grid_height * bin_grid_width);
  }
}

template<typename T>
__global__ void RoIAlignBackward(const int64_t nthreads, const T* out_diff_dptr,
                                 const float spatial_scale, const int32_t sampling_ratio,
                                 const int64_t channel_num, const int64_t height,
                                 const int64_t width, const int64_t roi_num,
                                 const int64_t pooled_height, const int64_t pooled_width,
                                 const T* rois_dptr, T* in_diff_dptr) {
  const int64_t pooled_area = pooled_height * pooled_width;
  const int64_t channel_pooled_area = channel_num * pooled_height * pooled_width;
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t h = (index / pooled_width) % pooled_height;
    const int64_t w = index % pooled_width;
    const int64_t c = (index / pooled_area) % channel_num;
    const int64_t r = (index / channel_pooled_area) % roi_num;
    const int64_t n = index / channel_pooled_area / roi_num;
    const T* offset_rois_dptr = rois_dptr + (n * roi_num + r) * 4;
    const T roi_start_w = offset_rois_dptr[0] * spatial_scale;
    const T roi_start_h = offset_rois_dptr[1] * spatial_scale;
    const T roi_end_w = offset_rois_dptr[2] * spatial_scale;
    const T roi_end_h = offset_rois_dptr[3] * spatial_scale;
    const T roi_height = max(roi_end_h - roi_start_h, static_cast<T>(1.0));
    const T roi_width = max(roi_end_w - roi_start_w, static_cast<T>(1.0));
    const T bin_height = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    const T bin_width = static_cast<T>(roi_width) / static_cast<T>(pooled_width);
    const int32_t bin_grid_height =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height);
    const int32_t bin_grid_width =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);
    const T bin_grid_density_h = bin_height / static_cast<T>(bin_grid_height);
    const T bin_grid_density_w = bin_width / static_cast<T>(bin_grid_width);

    T* in_diff_channel_dptr = in_diff_dptr + (n * channel_num + c) * height * width;
    const T* out_diff_channel_dptr = out_diff_dptr + (n * channel_num + c) * pooled_area;
    const T bin_diff = out_diff_channel_dptr[h * pooled_width + w];
    FOR_RANGE(int64_t, grid_i, 0, bin_grid_height) {
      // + .5f for center position
      const T y = roi_start_h + h * bin_height + static_cast<T>(grid_i + 0.5f) * bin_grid_density_h;
      FOR_RANGE(int64_t, grid_j, 0, bin_grid_width) {
        const T x =
            roi_start_w + w * bin_width + static_cast<T>(grid_j + 0.5f) * bin_grid_density_w;
        T diff11 = 0;
        T diff21 = 0;
        T diff12 = 0;
        T diff22 = 0;
        int32_t x_low = 0;
        int32_t x_high = 0;
        int32_t y_low = 0;
        int32_t y_high = 0;
        bool has_diff = BilinearInterpolateDiff(bin_diff / (bin_grid_height * bin_grid_width),
                                                height, width, y, x, diff11, diff21, diff12, diff22,
                                                x_low, x_high, y_low, y_high);
        if (has_diff) {
          const int64_t q11 = y_low * width + x_low;
          const int64_t q21 = y_low * width + x_high;
          const int64_t q12 = y_high * width + x_low;
          const int64_t q22 = y_high * width + x_high;
          gpu_atomic_add(in_diff_channel_dptr + q11, diff11);
          gpu_atomic_add(in_diff_channel_dptr + q21, diff21);
          gpu_atomic_add(in_diff_channel_dptr + q12, diff12);
          gpu_atomic_add(in_diff_channel_dptr + q22, diff22);
        }
      }
    }
  }
}
}  // namespace

template<typename T>
struct RoIAlignKernelUtil<DeviceType::kGPU, T> {
  static void Forward(const KernelCtx& ctx, const RoIAlignOpConf& conf, const Blob* in_blob,
                      const Blob* rois_blob, Blob* out_blob) {
    const int64_t elem_cnt = out_blob->shape().elem_cnt();
    RoIAlignForward<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                         ctx.device_ctx->hip_stream()>>>(
        elem_cnt, in_blob->dptr<T>(), conf.spatial_scale(), conf.sampling_ratio(),
        in_blob->shape().At(1), in_blob->shape().At(2), in_blob->shape().At(3),
        rois_blob->shape().At(1), conf.pooled_h(), conf.pooled_w(), rois_blob->dptr<T>(),
        out_blob->mut_dptr<T>());
  }

  static void Backward(const KernelCtx& ctx, const RoIAlignOpConf& conf, const Blob* out_diff_blob,
                       const Blob* rois_blob, Blob* in_diff_blob) {
    const int64_t elem_cnt = out_diff_blob->shape().elem_cnt();
    RoIAlignBackward<T><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                          ctx.device_ctx->hip_stream()>>>(
        elem_cnt, out_diff_blob->dptr<T>(), conf.spatial_scale(), conf.sampling_ratio(),
        in_diff_blob->shape().At(1), in_diff_blob->shape().At(2), in_diff_blob->shape().At(3),
        rois_blob->shape().At(1), conf.pooled_h(), conf.pooled_w(), rois_blob->dptr<T>(),
        in_diff_blob->mut_dptr<T>());
  }
};

#define INSTANTIATE_ROI_ALIGN_KERNEL_UTIL(type_cpp, type_proto) \
  template class RoIAlignKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_ROI_ALIGN_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ);

}  // namespace oneflow
