#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/adam_model_update_kernel.h"
#include "oneflow/core/kernel/normal_model_update_kernel.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void UpdateModelGpu(int64_t n, int64_t batch_size, T learning_rate, T l1, T l2, T beta1,
                               T beta2, T epsilon, bool do_bias_correction, const T* beta1_t,
                               const T* beta2_t, const T* model_diff, T* model, T* m, T* v) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    m[i] = beta1 * m[i] + (1 - beta1) * model_diff[i];
    v[i] = beta2 * v[i] + (1 - beta2) * model_diff[i] * model_diff[i];
    if (do_bias_correction) {
      learning_rate = learning_rate * sqrt(1 - (*beta2_t)) / (1 - (*beta1_t));
    }
    T reg_diff = RegularizeDiff(m[i] / (sqrt(v[i]) + epsilon), batch_size, l1, l2, model[i]);
    model[i] = model[i] - learning_rate * reg_diff;
  }
}

}  // namespace

template<typename T>
class AdamMdUpdateKernelUtil<DeviceType::kGPU, T> final {
 public:
  static void UpdateModel(DeviceCtx* ctx, int64_t n, int64_t batch_size, T learning_rate, T l1,
                          T l2, T beta1, T beta2, T epsilon, bool do_bias_correction,
                          const T* beta1_t, const T* beta2_t, const T* model_diff, T* model, T* m,
                          T* v) {
    UpdateModelGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, batch_size, learning_rate, l1, l2, beta1, beta2, epsilon, do_bias_correction, beta1_t,
        beta2_t, model_diff, model, m, v);
  }
};

#define INSTANTIATE_GPU_KERNEL_UTIL(type_cpp, type_proto) \
  template class AdamMdUpdateKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GPU_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
