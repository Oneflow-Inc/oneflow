#include <random>
#include "oneflow/core/device/cuda_device_context.h"
#include "oneflow/core/kernel/opkernel_test_common.h"

namespace oneflow {

#if defined(WITH_CUDA)

namespace test {

template<>
Blob* CreateBlob<DeviceType::kGPU>(const BlobDesc* blob_desc) {
  void* mem_ptr = nullptr;
  CudaCheck(hipMalloc(&mem_ptr, blob_desc->TotalByteSize()));
  return NewBlob(nullptr, blob_desc, static_cast<char*>(mem_ptr), nullptr,
                 DeviceType::kGPU);
}

template<>
void BuildKernelCtx<DeviceType::kGPU>(KernelCtx* ctx) {
  hipStream_t* hip_stream = new hipStream_t;
  hipblasHandle_t* cublas_pmh_handle = new hipblasHandle_t;
  hipblasHandle_t* cublas_pmd_handle = new hipblasHandle_t;
  hipdnnHandle_t* cudnn_handle = new hipdnnHandle_t;
  CudaCheck(hipStreamCreate(hip_stream));
  CudaCheck(hipblasCreate(cublas_pmh_handle));
  CudaCheck(hipblasCreate(cublas_pmd_handle));
  CudaCheck(hipblasSetStream(*cublas_pmh_handle, *hip_stream));
  CudaCheck(hipblasSetStream(*cublas_pmd_handle, *hip_stream));
  CudaCheck(
      hipblasSetPointerMode(*cublas_pmd_handle, HIPBLAS_POINTER_MODE_DEVICE));
  CudaCheck(hipdnnCreate(cudnn_handle));
  CudaCheck(hipdnnSetStream(*cudnn_handle, *hip_stream));
  ctx->device_ctx = new CudaDeviceCtx(-1, hip_stream, cublas_pmh_handle,
                                      cublas_pmd_handle, cudnn_handle, nullptr);
}

template<>
void SyncStream<DeviceType::kGPU>(KernelCtx* ctx) {
  CudaCheck(hipStreamSynchronize(ctx->device_ctx->hip_stream()));
}

template<typename T>
class KTCommon<DeviceType::kGPU, T> final {
 public:
  static void BlobCmp(const std::string& blob_name, const Blob* lhs,
                      const Blob* rhs) {
    Blob* cpu_lhs = CreateBlob<DeviceType::kCPU>(lhs->blob_desc_ptr());
    Blob* cpu_rhs = CreateBlob<DeviceType::kCPU>(rhs->blob_desc_ptr());
    CudaCheck(hipMemcpy(cpu_lhs->mut_dptr(), lhs->dptr(),
                         lhs->ByteSizeOfDataContentField(),
                         hipMemcpyDeviceToHost));
    CudaCheck(hipMemcpy(cpu_rhs->mut_dptr(), rhs->dptr(),
                         rhs->ByteSizeOfDataContentField(),
                         hipMemcpyDeviceToHost));
    KTCommon<DeviceType::kCPU, T>::BlobCmp(blob_name, cpu_lhs, cpu_rhs);
  }

  static void CheckInitializeResult(const Blob* blob,
                                    const InitializerConf& initializer_conf) {
    Blob* cpu_blob = CreateBlob<DeviceType::kCPU>(blob->blob_desc_ptr());
    CudaCheck(hipMemcpy(cpu_blob->mut_dptr(), blob->dptr(),
                         blob->ByteSizeOfDataContentField(),
                         hipMemcpyDeviceToHost));
    KTCommon<DeviceType::kCPU, T>::CheckInitializeResult(cpu_blob,
                                                         initializer_conf);
  }

 private:
  static Blob* CreateBlobWithSpecifiedValPtr(const BlobDesc* blob_desc,
                                             T* val) {
    Blob* ret = CreateBlob<DeviceType::kGPU>(blob_desc);
    CudaCheck(hipMemcpy(ret->mut_dptr(), val,
                         ret->ByteSizeOfDataContentField(),
                         hipMemcpyHostToDevice));
    return ret;
  }
};

#define INSTANTIATE_KTCOMMON(type_cpp, type_proto) \
  template class KTCommon<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KTCOMMON, ALL_DATA_TYPE_SEQ)

}  // namespace test

#endif

}  // namespace oneflow
