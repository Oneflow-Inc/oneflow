#include <random>
#include "oneflow/core/device/cuda_device_context.h"
#include "oneflow/core/kernel/opkernel_test_common.h"

namespace oneflow {

namespace test {

template<>
Blob* CreateBlob<DeviceType::kGPU>(const BlobDesc* blob_desc) {
  void* mem_ptr = nullptr;
  CudaCheck(hipMalloc(&mem_ptr, blob_desc->TotalByteSize()));
  return new Blob(nullptr, blob_desc, static_cast<char*>(mem_ptr));
}

template<>
void BuildKernelCtx<DeviceType::kGPU>(KernelCtx* ctx) {
  hipStream_t* hip_stream = new hipStream_t;
  hipblasHandle_t* cublas_handle = new hipblasHandle_t;
  CudaCheck(hipStreamCreate(hip_stream));
  CudaCheck(hipblasCreate(cublas_handle));
  CudaCheck(hipblasSetStream(*cublas_handle, *hip_stream));
  ctx->device_ctx = new CudaDeviceCtx(-1, hip_stream, cublas_handle, nullptr);
}

template<>
void SyncStream<DeviceType::kGPU>(KernelCtx* ctx) {
  CudaCheck(hipStreamSynchronize(ctx->device_ctx->hip_stream()));
}

template<typename T>
class KTCommon<DeviceType::kGPU, T> final {
 public:
  static Blob* CreateBlobWithSpecifiedVal(const BlobDesc* blob_desc, T* val) {
    Blob* ret = CreateBlob<DeviceType::kGPU>(blob_desc);
    CudaCheck(hipMemcpy(ret->mut_dptr(), val,
                         ret->ByteSizeOfDataContentField(),
                         hipMemcpyHostToDevice));
    return ret;
  }

  static void BlobCmp(const Blob* lhs, const Blob* rhs) {
    Blob* cpu_lhs = CreateBlob<DeviceType::kCPU>(lhs->blob_desc_ptr());
    Blob* cpu_rhs = CreateBlob<DeviceType::kCPU>(rhs->blob_desc_ptr());
    CudaCheck(hipMemcpy(cpu_lhs->mut_dptr(), lhs->dptr(),
                         lhs->ByteSizeOfDataContentField(),
                         hipMemcpyDeviceToHost));
    CudaCheck(hipMemcpy(cpu_rhs->mut_dptr(), rhs->dptr(),
                         rhs->ByteSizeOfDataContentField(),
                         hipMemcpyDeviceToHost));
    KTCommon<DeviceType::kCPU, T>::BlobCmp(cpu_lhs, cpu_rhs);
  }

  static void CheckInitializeResult(const Blob* blob,
                                    const InitializerConf& initializer_conf) {
    Blob* cpu_blob = CreateBlob<DeviceType::kCPU>(blob->blob_desc_ptr());
    CudaCheck(hipMemcpy(cpu_blob->mut_dptr(), blob->dptr(),
                         blob->ByteSizeOfDataContentField(),
                         hipMemcpyDeviceToHost));
    KTCommon<DeviceType::kCPU, T>::CheckInitializeResult(cpu_blob,
                                                         initializer_conf);
  }
};

#define INSTANTIATE_KTCOMMON(type_cpp, type_proto) \
  template class KTCommon<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KTCOMMON, ALL_DATA_TYPE_SEQ)

}  // namespace test

}  // namespace oneflow
