#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/batch_gather_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include <assert.h>

namespace oneflow {

namespace {

Shape GetFlatShape(const ShapeView& shape, const int64_t axis) {
  CHECK_GT(shape.NumAxes(), 0);
  CHECK_GE(axis, 0);
  CHECK_LT(axis, shape.NumAxes());
  return Shape({shape.Count(0, axis), shape.At(axis), shape.Count(axis + 1)});
}

template<typename K>
__device__ int64_t GetInOffset(const int64_t out_offset, const K* indices,
                               const int64_t indices_num, const int64_t instance_size,
                               const int64_t gather_dim_size) {
  const int64_t batch_idx = out_offset / (indices_num * instance_size);
  const int64_t indices_idx = out_offset % (indices_num * instance_size) / instance_size;
  const int64_t inner_idx = out_offset % instance_size;
  const int64_t idx = indices[batch_idx * indices_num + indices_idx];
  assert(idx >= 0 && idx < gather_dim_size);
  return batch_idx * gather_dim_size * instance_size + idx * instance_size + inner_idx;
}

template<typename T, typename K>
__global__ void BatchGatherForwardGpu(const int64_t elem_cnt, const T* in, const K* indices,
                                      const int64_t indices_num, const int64_t instance_size,
                                      const int64_t gather_dim_size, T* out) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    out[i] = in[GetInOffset<K>(i, indices, indices_num, instance_size, gather_dim_size)];
  }
}

template<typename T, typename K>
__global__ void BatchGatherBackwardGpu(const int64_t elem_cnt, const T* out_diff, const K* indices,
                                       const int64_t indices_num, const int64_t instance_size,
                                       const int64_t gather_dim_size, T* in_diff) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    gpu_atomic_add(
        in_diff + GetInOffset<K>(i, indices, indices_num, instance_size, gather_dim_size),
        out_diff[i]);
  }
}

}  // namespace

template<typename T, typename K>
struct BatchGatherKernelUtilImpl<DeviceType::kGPU, T, K> final {
  static void Forward(DeviceCtx* ctx, const T* in, const K* indices, const Shape& flat_out_shape,
                      const int64_t gather_dim_size, T* out);
  static void Backward(DeviceCtx* ctx, const T* out_diff, const K* indices,
                       const Shape& flat_out_diff_shape, const int64_t gather_dim_size, T* in_diff);
};

template<typename T, typename K>
void BatchGatherKernelUtilImpl<DeviceType::kGPU, T, K>::Forward(DeviceCtx* ctx, const T* in,
                                                                const K* indices,
                                                                const Shape& flat_out_shape,
                                                                const int64_t gather_dim_size,
                                                                T* out) {
  const int64_t batch_num = flat_out_shape.At(0);
  const int64_t indices_num = flat_out_shape.At(1);
  const int64_t instance_size = flat_out_shape.At(2);
  const int64_t elem_cnt = batch_num * indices_num * instance_size;
  BatchGatherForwardGpu<T, K>
      <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          elem_cnt, in, indices, indices_num, instance_size, gather_dim_size, out);
}

template<typename T, typename K>
void BatchGatherKernelUtilImpl<DeviceType::kGPU, T, K>::Backward(DeviceCtx* ctx, const T* out_diff,
                                                                 const K* indices,
                                                                 const Shape& flat_out_diff_shape,
                                                                 const int64_t gather_dim_size,
                                                                 T* in_diff) {
  const int64_t batch_num = flat_out_diff_shape.At(0);
  const int64_t indices_num = flat_out_diff_shape.At(1);
  const int64_t instance_size = flat_out_diff_shape.At(2);
  const int64_t elem_cnt = batch_num * indices_num * instance_size;
  BatchGatherBackwardGpu<T, K>
      <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          elem_cnt, out_diff, indices, indices_num, instance_size, gather_dim_size, in_diff);
}

#define INSTANTIATE_BATCH_GATHER_KERNEL_UTIL_IMPL_GPU(in_type_pair, index_type_pair)          \
  template struct BatchGatherKernelUtilImpl<DeviceType::kGPU, OF_PP_PAIR_FIRST(in_type_pair), \
                                            OF_PP_PAIR_FIRST(index_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_BATCH_GATHER_KERNEL_UTIL_IMPL_GPU,
                                 FLOATING_DATA_TYPE_SEQ, INT_DATA_TYPE_SEQ);
#undef INSTANTIATE_BATCH_GATHER_KERNEL_UTIL_IMPL_GPU

template<typename T>
class BatchGatherGPUKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(BatchGatherGPUKernel);
  BatchGatherGPUKernel() = default;
  ~BatchGatherGPUKernel() override = default;

 private:
  const PbMessage& GetCustomizedOpConf() const override {
    return this->op_conf().batch_gather_conf();
  }

  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    const Blob* in = BnInOp2Blob("in");
    const Blob* indices = BnInOp2Blob("indices");
    Blob* out = BnInOp2Blob("out");
    const int64_t axis = indices->shape().NumAxes() - 1;
    const Shape& flat_out_shape = GetFlatShape(out->shape(), axis);

    const int64_t batch_num = flat_out_shape.At(0);
    const int64_t indices_num = flat_out_shape.At(1);
    const int64_t instance_size = flat_out_shape.At(2);
    const int64_t elem_cnt = batch_num * indices_num * instance_size;
    BatchGatherForwardGpu<T, int32_t><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                        ctx.device_ctx->hip_stream()>>>(
        elem_cnt, in->dptr<T>(), indices->dptr<int32_t>(), indices_num, instance_size,
        in->shape().At(axis), out->mut_dptr<T>());
  }
};

REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kBatchGatherConf, DeviceType::kGPU, int32_t,
                                      BatchGatherGPUKernel<int32_t>)
REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kBatchGatherConf, DeviceType::kGPU, float,
                                      BatchGatherGPUKernel<float>)
REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kBatchGatherConf, DeviceType::kGPU, double,
                                      BatchGatherGPUKernel<double>)

template<typename T>
class UnsortedBatchSegmentSumGPUKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(UnsortedBatchSegmentSumGPUKernel);
  UnsortedBatchSegmentSumGPUKernel() = default;
  ~UnsortedBatchSegmentSumGPUKernel() override = default;

 private:
  const PbMessage& GetCustomizedOpConf() const override {
    return this->op_conf().unsorted_batch_segment_sum_conf();
  }
  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    const Blob* out_diff = BnInOp2Blob("data");
    const Blob* indices = BnInOp2Blob("segment_ids");
    Blob* in_diff = BnInOp2Blob("out");
    const int64_t axis = indices->shape().NumAxes() - 1;
    const Shape& flat_out_diff_shape = GetFlatShape(out_diff->shape(), axis);
    const int64_t batch_num = flat_out_diff_shape.At(0);
    const int64_t indices_num = flat_out_diff_shape.At(1);
    const int64_t instance_size = flat_out_diff_shape.At(2);
    const int64_t elem_cnt = batch_num * indices_num * instance_size;
    const int64_t gather_dim_size = in_diff->shape().At(axis);
    Memset<DeviceType::kGPU>(ctx.device_ctx, in_diff->mut_dptr<T>(), 0,
                             in_diff->ByteSizeOfBlobBody());
    BatchGatherBackwardGpu<T, int32_t><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                         ctx.device_ctx->hip_stream()>>>(
        elem_cnt, out_diff->dptr<T>(), indices->dptr<int32_t>(), indices_num, instance_size,
        gather_dim_size, in_diff->mut_dptr<T>());
  }
};

REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kUnsortedBatchSegmentSumConf, DeviceType::kGPU,
                                      int32_t, UnsortedBatchSegmentSumGPUKernel<int32_t>)
REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kUnsortedBatchSegmentSumConf, DeviceType::kGPU,
                                      float, UnsortedBatchSegmentSumGPUKernel<float>)
REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kUnsortedBatchSegmentSumConf, DeviceType::kGPU,
                                      double, UnsortedBatchSegmentSumGPUKernel<double>)

}  // namespace oneflow
