#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/lars_model_update_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void SumOfSquareGpu(const int64_t n, const T* x, T* result) {
  CUDA_1D_KERNEL_LOOP(i, n) { *result += x[i] * x[i]; }
}

}  // namespace

template<typename T>
class LARSMdUpdateKernelUtil<DeviceType::kGPU, T> final {
 public:
  static void SumOfSquare(DeviceCtx* ctx, const int64_t n, const T* x, T* result) {
    SumOfSquareGpu<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(n, x, result);
  }
};

#define INSTANTIATE_GPU_KERNEL_UTIL(type_cpp, type_proto) \
  template class LARSMdUpdateKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GPU_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
