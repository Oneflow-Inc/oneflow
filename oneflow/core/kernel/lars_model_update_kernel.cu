#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/lars_model_update_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void GetLocalLearningRateGpu(const float* learning_rate, T weight_decay, T epsilon,
                                        T lars_coefficient, const int64_t* train_step,
                                        T* data_tmp) {
  T* model_norm = &data_tmp[0];
  T* model_diff_norm = &data_tmp[1];
  T* local_learning_rate = &data_tmp[2];
  *model_norm = std::sqrt(*model_norm);
  *model_diff_norm = std::sqrt(*model_diff_norm);  // TODO(shiyuan)
  if (*train_step == 0) {
    *local_learning_rate =
        *learning_rate * lars_coefficient * (*model_norm) / (epsilon + (*model_diff_norm));
  } else {
    *local_learning_rate = *learning_rate * lars_coefficient * (*model_norm)
                           / (epsilon + (*model_diff_norm) + weight_decay * (*model_diff_norm));
  }
}

template<typename T>
__global__ void UpdateModelGpu(int64_t n, T weight_decay, T momentum_beta, const T* model_diff,
                               T* model, T* momentum, T* data_tmp) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    T reg_diff = model_diff[i] + weight_decay * model[i];
    momentum[i] = momentum_beta * momentum[i] - data_tmp[2] * reg_diff;
    model[i] = model[i] + momentum[i];
  }
}

}  // namespace

template<typename T>
class LARSMdUpdateKernelUtil<DeviceType::kGPU, T> final {
 public:
  static void UpdateModel(DeviceCtx* ctx, int64_t n, const float* learning_rate, T weight_decay,
                          T momentum_beta, T epsilon, T lars_coefficient, const int64_t* train_step,
                          const T* model_diff, T* model, T* momentum, T* data_tmp) {
    KernelUtil<DeviceType::kGPU, T>::Dot(ctx, n, model, 1, model, 1, &data_tmp[0]);
    KernelUtil<DeviceType::kGPU, T>::Dot(ctx, n, model_diff, 1, model_diff, 1, &data_tmp[1]);
    GetLocalLearningRateGpu<T><<<1, 1, 0, ctx->hip_stream()>>>(
        learning_rate, weight_decay, epsilon, lars_coefficient, train_step, data_tmp);
    UpdateModelGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, weight_decay, momentum_beta, model_diff, model, momentum, data_tmp);
  }
};

#define INSTANTIATE_GPU_KERNEL_UTIL(type_cpp, type_proto) \
  template class LARSMdUpdateKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GPU_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
