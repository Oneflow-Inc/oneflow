#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/lars_model_update_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void SumOfSquareGpu(int64_t n, const T* x, T* result) {
  CUDA_1D_KERNEL_LOOP(i, n) { *result += x[i] * x[i]; }
}

template<typename T>
__global__ void GetLocalLearningRateGpu(int64_t n, int64_t batch_size, T learning_rate, T l2,
                                        T epsilon, T lars_coefficient, int64_t next_model_vid,
                                        T* data_tmp) {
  T* model_norm = &data_tmp[0];
  T* model_diff_norm = &data_tmp[1];
  T* local_learning_rate = &data_tmp[2];
  *model_norm = std::sqrt(*model_norm / n);
  *model_diff_norm = std::sqrt(*model_diff_norm / n);
  if (next_model_vid == 1) {
    *local_learning_rate =
        learning_rate * lars_coefficient * (*model_norm) / (epsilon + (*model_diff_norm));
  } else {
    *local_learning_rate = learning_rate * lars_coefficient * (*model_norm)
                           / (epsilon + (*model_diff_norm) + l2 * (*model_norm));
  }
}

}  // namespace

template<typename T>
class LARSMdUpdateKernelUtil<DeviceType::kGPU, T> final {
 public:
  static void UpdateModel(DeviceCtx* ctx, int64_t n, int64_t batch_size, T learning_rate, T l1,
                          T l2, T momentum_beta, T epsilon, T lars_coefficient,
                          int64_t next_model_vid, const T* pre_model, const T* model_diff,
                          T* momentum, T* model, T* data_tmp) {
    SumOfSquareGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, pre_model, &data_tmp[0]);
    SumOfSquareGpu<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, model_diff, &data_tmp[1]);
    GetLocalLearningRateGpu<T>
        <<<BlocksNum4ThreadsNum(1), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            n, batch_size, learning_rate, l2, epsilon, lars_coefficient, next_model_vid, data_tmp);
    CudaCheck(hipStreamSynchronize(ctx->hip_stream()));
    T local_learning_rate;
    CudaCheck(hipMemcpy(&local_learning_rate, &data_tmp[2], sizeof(T), hipMemcpyDeviceToHost));
    NormalMdUpdateKernelUtil<DeviceType::kGPU, T>::UpdateModel(
        ctx, n, batch_size, local_learning_rate, l1, l2, momentum_beta, pre_model, model_diff,
        momentum, model);
  }
};

#define INSTANTIATE_GPU_KERNEL_UTIL(type_cpp, type_proto) \
  template class LARSMdUpdateKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GPU_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
