#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/convolution_kernel.h"
#include "oneflow/core/kernel/kernel_util.h"

namespace oneflow {

namespace {
template<typename FloatingPointType>
__global__ void Im2ColGpuKernel(const int n, const FloatingPointType* data_im,
                                const int height, const int width,
                                const int kernel_h, const int kernel_w,
                                const int pad_h, const int pad_w,
                                const int stride_h, const int stride_w,
                                const int dilation_h, const int dilation_w,
                                const int height_col, const int width_col,
                                FloatingPointType* data_col) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int h_index = index / width_col;
    const int h_col = h_index % height_col;
    const int w_col = index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;
    FloatingPointType* data_col_ptr = data_col;
    data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
    const FloatingPointType* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i * dilation_h;
        int w_im = w_offset + j * dilation_w;
        *data_col_ptr =
            (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width)
                ? data_im_ptr[i * dilation_h * width + j * dilation_w]
                : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template<typename FloatingPointType>
__global__ void Col2ImGpuKernel(
    const int n, const FloatingPointType* data_col, const int height,
    const int width, const int channels, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w, const int height_col,
    const int width_col, FloatingPointType* data_im) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    FloatingPointType val = 0;
    const int w_im = index % width + pad_w;
    const int h_im = (index / width) % height + pad_h;
    const int c_im = index / (width * height);
    int kernel_extent_w = (kernel_w - 1) * dilation_w + 1;
    int kernel_extent_h = (kernel_h - 1) * dilation_h + 1;
    // compute the start and end of the output
    const int w_col_start =
        (w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) / stride_w + 1;
    const int w_col_end = min(w_im / stride_w + 1, width_col);
    const int h_col_start =
        (h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) / stride_h + 1;
    const int h_col_end = min(h_im / stride_h + 1, height_col);
    // TODO: use LCM of stride and dilation to avoid unnecessary loops
    for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
      for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
        int h_k = (h_im - h_col * stride_h);
        int w_k = (w_im - w_col * stride_w);
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          int data_col_index =
              (((c_im * kernel_h + h_k) * kernel_w + w_k) * height_col + h_col)
                  * width_col
              + w_col;
          val += data_col[data_col_index];
        }
      }
    }
    data_im[index] = val;
  }
}

}  // namespace

template<typename FloatingPointType>
class ConvolutionKernelUtil<DeviceType::kGPU, FloatingPointType> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(ConvolutionKernelUtil);
  ConvolutionKernelUtil() = delete;
  static void Im2Col(const KernelCtx& ctx, const FloatingPointType* data_im,
                     const int channels, const int height, const int width,
                     const int kernel_h, const int kernel_w, const int pad_h,
                     const int pad_w, const int stride_h, const int stride_w,
                     const int dilation_h, const int dilation_w,
                     FloatingPointType* data_col) {
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col =
        (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    int width_col =
        (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    int num_kernels = channels * height_col * width_col;
    Im2ColGpuKernel<FloatingPointType>
        <<<BlocksNum4ThreadsNum(num_kernels), kCudaThreadsNumPerBlock, 0,
           ctx.device_ctx->hip_stream()>>>(
            num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h,
            pad_w, stride_h, stride_w, dilation_h, dilation_w, height_col,
            width_col, data_col);
  }

  static void Col2Im(const KernelCtx& ctx, const FloatingPointType* data_col,
                     const int channels, const int height, const int width,
                     const int kernel_h, const int kernel_w, const int pad_h,
                     const int pad_w, const int stride_h, const int stride_w,
                     const int dilation_h, const int dilation_w,
                     FloatingPointType* data_im) {
    int height_col =
        (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    int width_col =
        (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    int num_kernels = channels * height * width;
    // To avoid involving atomic operations, we will launch one kernel per
    // bottom dimension, and then in the kernel add up the top dimensions.
    Col2ImGpuKernel<FloatingPointType>
        <<<BlocksNum4ThreadsNum(num_kernels), kCudaThreadsNumPerBlock, 0,
           ctx.device_ctx->hip_stream()>>>(
            num_kernels, data_col, height, width, channels, kernel_h, kernel_w,
            pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
            height_col, width_col, data_im);
  }
};

INSTANTIATE_GPU_KERNEL_UTIL_CLASS(ConvolutionKernelUtil);
}  // namespace oneflow
