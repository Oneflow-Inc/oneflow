#include "oneflow/core/kernel/copy_hd_kernel.h"
#include <string>
#include "oneflow/core/operator/op_conf.pb.h"

namespace oneflow {

template<typename floating_point_type>
void CopyHdKernel<DeviceType::kGPU, floating_point_type>::Forward(
    const KernelContext& ctx,
    std::function<Blob*(const std::string&)> BnInOp2BlobPtr) const {
  const std::string& ibn = op()->SoleIbn();
  Blob* in_blob = BnInOp2BlobPtr(ibn);
  const std::string& obn = op()->SoleObn();
  Blob* out_blob = BnInOp2BlobPtr(obn);
  
  const CopyHdOpConf& copy_hd_conf = op()->op_conf().copy_hd_conf();
  
  if (copy_hd_conf.type() == copy_hd_conf.H2D) {
    hipMemcpyAsync(out_blob->mut_dptr(), in_blob->dptr(), 
                    in_blob->shape().elem_cnt()*sizeof(floating_point_type),
                    hipMemcpyHostToDevice,
                    *(ctx.hip_stream));
  } else {
    hipMemcpyAsync(out_blob->mut_dptr(), in_blob->dptr(), 
                    in_blob->shape().elem_cnt()*sizeof(floating_point_type),
                    hipMemcpyDeviceToHost, 
                    *(ctx.hip_stream));
  }
}

template<typename floating_point_type>
void CopyHdKernel<DeviceType::kGPU, floating_point_type>::Backward(
    const KernelContext& ctx,
    std::function<Blob*(const std::string&)> BnInOp2BlobPtr) const {
  const std::string& odbn = op()->SoleOdbn();
  Blob* in_blob = BnInOp2BlobPtr(odbn);
  const std::string& idbn = op()->SoleIdbn();
  Blob* out_blob = BnInOp2BlobPtr(idbn);

  const CopyHdOpConf& copy_hd_conf = op()->op_conf().copy_hd_conf();

  if (copy_hd_conf.type() == copy_hd_conf.H2D) {
    hipMemcpyAsync(out_blob->mut_dptr(), in_blob->dptr(), 
                    in_blob->shape().elem_cnt()*sizeof(floating_point_type),
                    hipMemcpyHostToDevice,
                    *(ctx.hip_stream));
  } else {
    hipMemcpyAsync(out_blob->mut_dptr(), in_blob->dptr(), 
                    in_blob->shape().elem_cnt()*sizeof(floating_point_type),
                    hipMemcpyDeviceToHost,
                    *(ctx.hip_stream));
  }
}

INSTANTIATE_GPU_KERNEL_CLASS(CopyHdKernel);
REGISTER_KERNEL(OperatorConf::kCopyHdConf, CopyHdKernel);

}  // namespace oneflow
