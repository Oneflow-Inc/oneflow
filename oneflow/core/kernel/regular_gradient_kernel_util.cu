#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/regular_gradient_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void RegularGradientGpu(int64_t n, const T* model, const T* model_diff, T* out,
                                   const T l1_scale, const T l2_scale) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    const T model_val = model[i];
    out[i] =
        model_diff[i] + l1_scale * ((model_val >= 0) - (model_val <= 0)) + l2_scale * model_val;
  }
}

}  // namespace

template<typename T>
struct RegularGradientKernelUtil<DeviceType::kGPU, T> {
  static void RegularGradient(DeviceCtx* ctx, int64_t n, const T* model, const T* model_diff,
                              T* out, const T l1_scale, const T l2_scale) {
    RegularGradientGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        n, model, model_diff, out, l1_scale, l2_scale);
  }
};

#define INSTANTIATE_REGULAR_GRADIENT_KERNEL_UTIL_GPU(type_cpp, type_proto) \
  template struct RegularGradientKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_REGULAR_GRADIENT_KERNEL_UTIL_GPU, FLOATING_DATA_TYPE_SEQ);
#undef INSTANTIATE_REGULAR_GRADIENT_KERNEL_UTIL_GPU

}  // namespace oneflow
