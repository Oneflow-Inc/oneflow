#include "oneflow/core/kernel/opkernel_test_case.h"
#include <random>
#include "oneflow/core/device/cuda_device_context.h"

namespace oneflow {

namespace test {

#if defined(WITH_CUDA)

template<>
Blob* CreateBlob<DeviceType::kGPU>(const BlobDesc* blob_desc) {
  void* mem_ptr = nullptr;
  CudaCheck(hipMalloc(&mem_ptr, blob_desc->TotalByteSize()));
  return NewBlob(nullptr, blob_desc, static_cast<char*>(mem_ptr), nullptr,
                 DeviceType::kGPU);
}

template<>
void BuildKernelCtx<DeviceType::kGPU>(KernelCtx* ctx) {
  hipStream_t* hip_stream = new hipStream_t;
  hipblasHandle_t* cublas_pmh_handle = new hipblasHandle_t;
  hipblasHandle_t* cublas_pmd_handle = new hipblasHandle_t;
  hipdnnHandle_t* cudnn_handle = new hipdnnHandle_t;
  CudaCheck(hipStreamCreate(hip_stream));
  CudaCheck(hipblasCreate(cublas_pmh_handle));
  CudaCheck(hipblasCreate(cublas_pmd_handle));
  CudaCheck(hipblasSetStream(*cublas_pmh_handle, *hip_stream));
  CudaCheck(hipblasSetStream(*cublas_pmd_handle, *hip_stream));
  CudaCheck(
      hipblasSetPointerMode(*cublas_pmd_handle, HIPBLAS_POINTER_MODE_DEVICE));
  CudaCheck(hipdnnCreate(cudnn_handle));
  CudaCheck(hipdnnSetStream(*cudnn_handle, *hip_stream));
  ctx->device_ctx = new CudaDeviceCtx(-1, hip_stream, cublas_pmh_handle,
                                      cublas_pmd_handle, cudnn_handle, nullptr);
}

template<>
void SyncStream<DeviceType::kGPU>(KernelCtx* ctx) {
  CudaCheck(hipStreamSynchronize(ctx->device_ctx->hip_stream()));
}

template<>
template<typename T>
Blob* OpKernelTestCase<DeviceType::kGPU>::CreateBlobWithSpecifiedValPtr(
    const BlobDesc* blob_desc, T* val) const {
  Blob* ret = CreateBlob<DeviceType::kGPU>(blob_desc);
  CudaCheck(hipMemcpy(ret->mut_dptr(), val, ret->ByteSizeOfDataContentField(),
                       hipMemcpyHostToDevice));
  return ret;
}

template<>
template<typename T>
void OpKernelTestCase<DeviceType::kGPU>::BlobCmp(const std::string& blob_name,
                                                 const Blob* lhs,
                                                 const Blob* rhs) const {
  Blob* cpu_lhs = CreateBlob<DeviceType::kCPU>(lhs->blob_desc_ptr());
  Blob* cpu_rhs = CreateBlob<DeviceType::kCPU>(rhs->blob_desc_ptr());
  CudaCheck(hipMemcpy(cpu_lhs->mut_dptr(), lhs->dptr(),
                       lhs->ByteSizeOfDataContentField(),
                       hipMemcpyDeviceToHost));
  CudaCheck(hipMemcpy(cpu_rhs->mut_dptr(), rhs->dptr(),
                       rhs->ByteSizeOfDataContentField(),
                       hipMemcpyDeviceToHost));
  OpKernelTestCase<DeviceType::kCPU>().template BlobCmp<T>(blob_name, cpu_lhs,
                                                           cpu_rhs);
}

#define INSTANTIATE_METHODS(type_cpp, type_proto)                      \
  template Blob*                                                       \
  OpKernelTestCase<DeviceType::kGPU>::CreateBlobWithSpecifiedValPtr(   \
      const BlobDesc* blob_desc, type_cpp* val) const;                 \
  template void OpKernelTestCase<DeviceType::kGPU>::BlobCmp<type_cpp>( \
      const std::string& blob_name, const Blob* lhs, const Blob* rhs) const;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_METHODS, ALL_DATA_TYPE_SEQ);

template<typename T>
class KTCommon<DeviceType::kGPU, T> final {
 public:
  static void CheckInitializeResult(const Blob* blob,
                                    const InitializerConf& initializer_conf) {
    Blob* cpu_blob = CreateBlob<DeviceType::kCPU>(blob->blob_desc_ptr());
    CudaCheck(hipMemcpy(cpu_blob->mut_dptr(), blob->dptr(),
                         blob->ByteSizeOfDataContentField(),
                         hipMemcpyDeviceToHost));
    KTCommon<DeviceType::kCPU, T>::CheckInitializeResult(cpu_blob,
                                                         initializer_conf);
  }
};

#define INSTANTIATE_KTCOMMON(type_cpp, type_proto) \
  template class KTCommon<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_KTCOMMON, ALL_DATA_TYPE_SEQ)

#endif

}  // namespace test

}  // namespace oneflow
