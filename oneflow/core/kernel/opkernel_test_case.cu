#include "oneflow/core/kernel/opkernel_test_case.h"
#include <random>
#include "oneflow/core/device/cuda_device_context.h"

namespace oneflow {

namespace test {

#if defined(WITH_CUDA)

template<>
Blob* OpKernelTestCase<DeviceType::kGPU>::CreateBlob(const BlobDesc* blob_desc,
                                                     Regst* regst) {
  void* mem_ptr = nullptr;
  CudaCheck(hipMalloc(&mem_ptr, blob_desc->TotalByteSize()));
  return NewBlob(regst, blob_desc, static_cast<char*>(mem_ptr), nullptr,
                 DeviceType::kGPU);
}

template<>
void OpKernelTestCase<DeviceType::kGPU>::BuildKernelCtx(KernelCtx* ctx) {
  hipStream_t* hip_stream = new hipStream_t;
  hipblasHandle_t* cublas_pmh_handle = new hipblasHandle_t;
  hipblasHandle_t* cublas_pmd_handle = new hipblasHandle_t;
  hipdnnHandle_t* cudnn_handle = new hipdnnHandle_t;
  CudaCheck(hipStreamCreate(hip_stream));
  CudaCheck(hipblasCreate(cublas_pmh_handle));
  CudaCheck(hipblasCreate(cublas_pmd_handle));
  CudaCheck(hipblasSetStream(*cublas_pmh_handle, *hip_stream));
  CudaCheck(hipblasSetStream(*cublas_pmd_handle, *hip_stream));
  CudaCheck(
      hipblasSetPointerMode(*cublas_pmd_handle, HIPBLAS_POINTER_MODE_DEVICE));
  CudaCheck(hipdnnCreate(cudnn_handle));
  CudaCheck(hipdnnSetStream(*cudnn_handle, *hip_stream));
  ctx->device_ctx = new CudaDeviceCtx(-1, hip_stream, cublas_pmh_handle,
                                      cublas_pmd_handle, cudnn_handle, nullptr);
}

template<>
void OpKernelTestCase<DeviceType::kGPU>::SyncStream(KernelCtx* ctx) {
  CudaCheck(hipStreamSynchronize(ctx->device_ctx->hip_stream()));
}

template<>
template<typename T>
Blob* OpKernelTestCase<DeviceType::kGPU>::CreateBlobWithSpecifiedValPtr(
    const BlobDesc* blob_desc, T* val, Regst* regst) {
  Blob* ret = CreateBlob(blob_desc, regst);
  CudaCheck(hipMemcpy(ret->mut_dptr(), val, ret->ByteSizeOfDataContentField(),
                       hipMemcpyHostToDevice));
  return ret;
}

template<>
template<typename T>
void OpKernelTestCase<DeviceType::kGPU>::BlobCmp(const std::string& blob_name,
                                                 const Blob* lhs,
                                                 const Blob* rhs) {
  Blob* cpu_lhs = OpKernelTestCase<DeviceType::kCPU>::CreateBlob(
      lhs->blob_desc_ptr(), nullptr);
  Blob* cpu_rhs = OpKernelTestCase<DeviceType::kCPU>::CreateBlob(
      rhs->blob_desc_ptr(), nullptr);
  CudaCheck(hipMemcpy(cpu_lhs->mut_dptr(), lhs->dptr(),
                       lhs->ByteSizeOfDataContentField(),
                       hipMemcpyDeviceToHost));
  CudaCheck(hipMemcpy(cpu_rhs->mut_dptr(), rhs->dptr(),
                       rhs->ByteSizeOfDataContentField(),
                       hipMemcpyDeviceToHost));
  OpKernelTestCase<DeviceType::kCPU>::template BlobCmp<T>(blob_name, cpu_lhs,
                                                          cpu_rhs);
}

template<>
template<typename T>
void OpKernelTestCase<DeviceType::kGPU>::CheckInitializeResult(
    const Blob* blob, const InitializerConf& initializer_conf) {
  Blob* cpu_blob = OpKernelTestCase<DeviceType::kCPU>::CreateBlob(
      blob->blob_desc_ptr(), nullptr);
  CudaCheck(hipMemcpy(cpu_blob->mut_dptr(), blob->dptr(),
                       blob->ByteSizeOfDataContentField(),
                       hipMemcpyDeviceToHost));
  OpKernelTestCase<DeviceType::kCPU>::template CheckInitializeResult<T>(
      cpu_blob, initializer_conf);
}

#define INSTANTIATE_METHODS(type_cpp, type_proto)                      \
  template Blob*                                                       \
  OpKernelTestCase<DeviceType::kGPU>::CreateBlobWithSpecifiedValPtr(   \
      const BlobDesc* blob_desc, type_cpp* val, Regst* regst);         \
  template void OpKernelTestCase<DeviceType::kGPU>::BlobCmp<type_cpp>( \
      const std::string& blob_name, const Blob* lhs, const Blob* rhs); \
  template void                                                        \
  OpKernelTestCase<DeviceType::kGPU>::CheckInitializeResult<type_cpp>( \
      const Blob* blob, const InitializerConf& initializer_conf);
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_METHODS, ALL_DATA_TYPE_SEQ);

#endif

}  // namespace test

}  // namespace oneflow
