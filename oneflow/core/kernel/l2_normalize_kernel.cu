#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/l2_normalize_kernel.h"
#include <hipcub/hipcub.hpp>
#include <math.h>

namespace oneflow {

namespace {

template<typename T>
__global__ void L2NormalizeForward(const int32_t n, const int32_t c, const int32_t d,
                                   const T epsilon, const T* in, T* out) {
  using BlockReduce = hipcub::BlockReduce<T, kCudaThreadsNumPerBlock>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  for (int32_t i = blockIdx.x; i < n; i += gridDim.x) {
    T sum = ZeroVal<T>::value;
    int32_t beg = (i / d) * d * c + (i % d);
    for (int32_t j = threadIdx.x; j < c; j += blockDim.x) {
      const T x = in[beg + j * d];
      sum += x * x;
    }
    T reduce_sum = BlockReduce(temp_storage).Sum(sum);

    __shared__ T norm;
    if (threadIdx.x == 0) {
      norm = std::sqrt(reduce_sum);
      norm = fmaxf(norm, epsilon);
    }
    __syncthreads();

    for (int32_t j = threadIdx.x; j < c; j += blockDim.x) {
      const int32_t index = beg + j * d;
      out[index] = in[index] / norm;
    }
  }
}

template<typename T>
__global__ void L2NormalizeBackward(const int32_t n, const int32_t c, const int32_t d,
                                    const T epsilon, const T* in, const T* out_diff, T* in_diff) {
  using BlockReduce = hipcub::BlockReduce<T, kCudaThreadsNumPerBlock>;
  __shared__ typename BlockReduce::TempStorage temp_storage_squa_sum;
  __shared__ typename BlockReduce::TempStorage temp_storage_prod_sum;

  for (int32_t i = blockIdx.x; i < n; i += gridDim.x) {
    T squa_x_sum = ZeroVal<T>::value;
    T dy_x_prod_sum = ZeroVal<T>::value;
    int32_t beg = (i / d) * d * c + (i % d);
    for (int32_t j = threadIdx.x; j < c; j += blockDim.x) {
      const int32_t index = beg + j * d;
      squa_x_sum += in[index] * in[index];
      dy_x_prod_sum += in[index] * out_diff[index];
    }
    T reduce_squa_x_sum = BlockReduce(temp_storage_squa_sum).Sum(squa_x_sum);
    T reduce_dy_x_prod_sum = BlockReduce(temp_storage_prod_sum).Sum(dy_x_prod_sum);

    __shared__ T norm;
    __shared__ T norm_pow_3;
    __shared__ T xdy_sum;
    if (threadIdx.x == 0) {
      norm = std::sqrt(reduce_squa_x_sum);
      norm = fmaxf(norm, epsilon);
      norm_pow_3 = std::pow(norm, 3);
      xdy_sum = reduce_dy_x_prod_sum;
    }
    __syncthreads();

    for (int32_t j = threadIdx.x; j < c; j += blockDim.x) {
      const int32_t index = beg + j * d;
      const float x = in[index];
      const float dy = out_diff[index];
      in_diff[index] = (dy / norm) - ((x / norm_pow_3) * xdy_sum);
    }
  }
}

}  // namespace

template<typename T>
struct L2NormalizeKernelUtil<DeviceType::kGPU, T> {
  static void Forward(DeviceCtx* ctx, const L2NormalizeOpConf& conf, const Blob* in_blob,
                      Blob* out_blob) {
    int32_t axis = conf.axis() >= 0 ? conf.axis() : conf.axis() + in_blob->shape().NumAxes();
    int32_t c = in_blob->shape().At(axis);
    int32_t n = in_blob->shape().elem_cnt() / c;
    int32_t d = in_blob->shape().elem_cnt() / in_blob->shape().Count(0, axis + 1);
    L2NormalizeForward<<<std::min(n, kCudaMaxBlocksNum), kCudaThreadsNumPerBlock, 0,
                         ctx->hip_stream()>>>(n, c, d, static_cast<T>(conf.epsilon()),
                                               in_blob->dptr<T>(), out_blob->mut_dptr<T>());
  }

  static void Backward(DeviceCtx* ctx, const L2NormalizeOpConf& conf, const Blob* in_blob,
                       const Blob* out_diff_blob, Blob* in_diff_blob) {
    int32_t axis = conf.axis() >= 0 ? conf.axis() : conf.axis() + in_blob->shape().NumAxes();
    int32_t c = in_blob->shape().At(axis);
    int32_t n = in_blob->shape().elem_cnt() / c;
    int32_t d = in_blob->shape().elem_cnt() / in_blob->shape().Count(0, axis + 1);
    L2NormalizeBackward<<<std::min(n, kCudaMaxBlocksNum), kCudaThreadsNumPerBlock, 0,
                          ctx->hip_stream()>>>(n, c, d, static_cast<T>(conf.epsilon()),
                                                in_blob->dptr<T>(), out_diff_blob->dptr<T>(),
                                                in_diff_blob->mut_dptr<T>());
  }
};

#define INSTANTIATE_L2_NORMALIZE_KERNEL_UTIL(type_cpp, type_proto) \
  template struct L2NormalizeKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_L2_NORMALIZE_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
