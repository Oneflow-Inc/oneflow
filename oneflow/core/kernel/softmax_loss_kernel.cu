#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/kernel_manager.h"
#include "oneflow/core/kernel/softmax_kernel.h"
#include "oneflow/core/kernel/softmax_loss_kernel.h"

namespace oneflow {

namespace {

template<typename FloatingPointType>
__global__ void SoftmaxLossForwardTmp(const int64_t n, const int64_t w,
                                      const FloatingPointType* label,
                                      const FloatingPointType* prob,
                                      FloatingPointType* tmp) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    tmp[i] = -SAFE_LOG(prob[i * w + static_cast<int64_t>(label[i])]);
  }
}

template<typename FloatingPointType>
__global__ void SoftmaxLossBackwardSub(const int64_t n, const int64_t w,
                                       const FloatingPointType* label,
                                       FloatingPointType* in_diff) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    in_diff[i * w + static_cast<int64_t>(label[i])] -= 1;
  }
}

}  // namespace

template<typename FloatingPointType>
class SoftmaxLossKernelUtil<DeviceType::kGPU, FloatingPointType> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(SoftmaxLossKernelUtil);
  SoftmaxLossKernelUtil() = delete;

  static void ComputeLoss(const KernelCtx& ctx, const int64_t n,
                          const int64_t w, const FloatingPointType* label,
                          const FloatingPointType* prob, FloatingPointType* tmp,
                          FloatingPointType* loss) {
    SoftmaxLossForwardTmp<FloatingPointType>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx.device_ctx->hip_stream()>>>(n, w, label, prob, tmp);
    KernelUtil<DeviceType::kGPU, FloatingPointType>::Sum(
        ctx, n, tmp, loss, tmp, sizeof(FloatingPointType) * n);
  }

  static void BackwardSub(const KernelCtx& ctx, const int64_t n,
                          const int64_t w, const FloatingPointType* label,
                          FloatingPointType* in_diff) {
    SoftmaxLossBackwardSub<FloatingPointType>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx.device_ctx->hip_stream()>>>(n, w, label, in_diff);
  }
};

INSTANTIATE_GPU_KERNEL_UTIL_CLASS(SoftmaxLossKernelUtil);

}  // namespace oneflow
