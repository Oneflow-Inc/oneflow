#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/kernel_manager.h"
#include "oneflow/core/kernel/softmax_kernel.h"
#include "oneflow/core/kernel/softmax_loss_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void SoftmaxLossForwardTmp(const int64_t n, const int64_t w,
                                      const int32_t* label, const T* prob,
                                      T* tmp) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    tmp[i] = -SAFE_LOG(prob[i * w + static_cast<int64_t>(label[i])]);
  }
}

template<typename T>
__global__ void SoftmaxLossBackwardSub(const int64_t n, const int64_t w,
                                       const int32_t* label, T* in_diff) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    in_diff[i * w + static_cast<int64_t>(label[i])] -= 1;
  }
}

}  // namespace

template<typename T>
class SoftmaxLossKernelUtil<DeviceType::kGPU, T> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(SoftmaxLossKernelUtil);
  SoftmaxLossKernelUtil() = delete;

  static void ComputeLoss(DeviceCtx* ctx, const int64_t n, const int64_t w,
                          const int32_t* label, const T* prob, T* tmp,
                          T* loss) {
    SoftmaxLossForwardTmp<T><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock,
                               0, ctx->hip_stream()>>>(n, w, label, prob, tmp);
    KernelUtil<DeviceType::kGPU, T>::Sum(ctx, n, tmp, loss, tmp, sizeof(T) * n);
  }

  static void BackwardSub(DeviceCtx* ctx, const int64_t n, const int64_t w,
                          const int32_t* label, T* in_diff) {
    SoftmaxLossBackwardSub<T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,
           ctx->hip_stream()>>>(n, w, label, in_diff);
  }
};

INSTANTIATE_GPU_KERNEL_UTIL_FLOATING_TYPE(SoftmaxLossKernelUtil);

}  // namespace oneflow
