#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/device/nccl_util.h"
#include "nccl.h"

namespace oneflow {

class NcclTupleReduceKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(NcclTupleReduceKernel);
  NcclTupleReduceKernel() = default;
  ~NcclTupleReduceKernel() override = default;

 private:
  void VirtualKernelInit() override {}
  void ForwardDataContent(const KernelCtx&,
                          std::function<Blob*(const std::string&)>) const override;
};

void NcclTupleReduceKernel::ForwardDataContent(
    const KernelCtx& ctx, std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  const NcclTupleReduceOpConf& conf = this->op_conf().nccl_tuple_reduce_conf();
  const auto& parallel_ctx = this->kernel_conf().nccl_tuple_reduce_conf().parallel_ctx();
  NcclCheck(ncclGroupStart());
  FOR_RANGE(int64_t, i, 0, conf.out_size()) {
    const Blob* in = BnInOp2Blob(GenRepeatedBn("in", i));
    const void* send = in->dptr();
    Blob* out = BnInOp2Blob(GenRepeatedBn("out", i));
    void* recv = conf.root(i) == parallel_ctx.rank_ctx().rank_id() ? out->mut_dptr() : nullptr;
    NcclCheck(ncclReduce(send, recv, in->shape().elem_cnt(), GetNcclDataType(in->data_type()),
                         ncclRedOp_t::ncclSum, conf.root(i), ctx.device_ctx->nccl_handle(),
                         ctx.device_ctx->hip_stream()));
  }
  NcclCheck(ncclGroupEnd());
}

REGISTER_KERNEL_WITH_DEVICE(OperatorConf::kNcclTupleReduceConf, DeviceType::kGPU,
                            NcclTupleReduceKernel);

}  // namespace oneflow
