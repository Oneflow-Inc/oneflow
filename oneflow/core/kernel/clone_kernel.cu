#include "oneflow/core/kernel/clone_kernel.h"

namespace oneflow {

template<typename floating_point_type>
void CloneKernel<DeviceType::kGPU, floating_point_type>::Forward(
    const KernelCtx& ctx,
    std::function<Blob*(const std::string&)> BnInOp2BlobPtr) const {
  const Blob* in_blob = BnInOp2BlobPtr(op()->SoleIbn());
  for(const std::string& obn : op()->output_bns()) {
    Blob* out_blob = BnInOp2BlobPtr(obn);
    CHECK_EQ(hipMemcpyAsync(out_blob->mut_dptr(),
                             in_blob->dptr(),
                             in_blob->shape().elem_cnt() * sizeof(floating_point_type),
                             hipMemcpyDeviceToDevice,
                             ctx.device_ctx->hip_stream()),
             hipSuccess);
  }
}

template<typename floating_point_type>
void CloneKernel<DeviceType::kGPU, floating_point_type>::Backward(
    const KernelCtx& ctx,
    std::function<Blob*(const std::string&)> BnInOp2BlobPtr) const {
  Blob* idbn_blob = BnInOp2BlobPtr(op()->SoleIdbn());
  const std::vector<std::string>& odbns = op()->output_diff_bns();
  if (odbns.size() == 0) return;
  CHECK_EQ(hipMemcpyAsync(idbn_blob->mut_dptr(),
                           BnInOp2BlobPtr(odbns[0])->dptr(),
                           idbn_blob->shape().elem_cnt() * sizeof(floating_point_type),
                           hipMemcpyDeviceToDevice,
                           ctx.device_ctx->hip_stream()),
           hipSuccess);
  const floating_point_type alpha = {1.0f};
  for(size_t i = 1; i != odbns.size(); ++i) {
    const Blob* out_blob = BnInOp2BlobPtr(odbns[i]);
    cublas_axpy<floating_point_type>(
        ctx.device_ctx->cublas_handle(),
        idbn_blob->shape().elem_cnt(), &alpha,
        static_cast<const floating_point_type*>(out_blob->dptr()), 1,
        static_cast<floating_point_type*>(idbn_blob->mut_dptr()), 1);
  }
}

INSTANTIATE_GPU_KERNEL_CLASS(CloneKernel);
REGISTER_GPU_KERNEL(OperatorConf::kCloneConf, CloneKernel);

}  // namespace oneflow
