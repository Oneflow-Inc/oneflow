#include "oneflow/core/kernel/clone_kernel.h"
#include <string>
#include <typeinfo>

namespace oneflow {

namespace {

template<typename T>
hipblasStatus_t cublas_axpy(hipblasHandle_t handle, int n,
                           const T *alpha,
                           const T *x, int incx,
                           T *y, int incy){
  LOG(FATAL) << "floating_point_type should be flaot or double";
}

template<>
hipblasStatus_t cublas_axpy<float>(hipblasHandle_t handle, int n,
                                  const float *alpha,
                                  const float *x, int incx,
                                  float *y, int incy){
  return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}

template<>
hipblasStatus_t cublas_axpy<double>(hipblasHandle_t handle, int n,
                                   const double *alpha,
                                   const double *x, int incx,
                                   double *y, int incy){
  return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}

} // namespace

template<typename floating_point_type>
void CloneKernel<DeviceType::kGPU, floating_point_type>::Forward(
    const KernelCtx& ctx,
    std::function<Blob*(const std::string&)> BnInOp2BlobPtr) const {
  const Blob* in_blob = BnInOp2BlobPtr(op()->SoleIbn());
  for(const std::string& obn : op()->output_bns()) {
    Blob* out_blob = BnInOp2BlobPtr(obn);
    CHECK_EQ(hipMemcpyAsync(out_blob->mut_dptr(),
                             in_blob->dptr(),
                             in_blob->shape().elem_cnt() * sizeof(floating_point_type),
                             hipMemcpyDeviceToDevice,
                             ctx.device_ctx->hip_stream()),
             hipSuccess);
  }
}

template<typename floating_point_type>
void CloneKernel<DeviceType::kGPU, floating_point_type>::Backward(
    const KernelCtx& ctx,
    std::function<Blob*(const std::string&)> BnInOp2BlobPtr) const {
  Blob* idbn_blob = BnInOp2BlobPtr(op()->SoleIdbn());
  const std::vector<std::string>& odbns = op()->output_diff_bns();
  if (odbns.size() == 0) return;
  CHECK_EQ(hipMemcpyAsync(idbn_blob->mut_dptr(),
                           BnInOp2BlobPtr(odbns[0])->dptr(),
                           idbn_blob->shape().elem_cnt() * sizeof(floating_point_type),
                           hipMemcpyDeviceToDevice,
                           ctx.device_ctx->hip_stream()),
           hipSuccess);
  const floating_point_type alpha = {1.0f};
  for(size_t i = 1; i != odbns.size(); ++i) {
    const Blob* odbn_blob = BnInOp2BlobPtr(odbns[i]);
    CHECK_EQ(cublas_axpy<floating_point_type>(
                 ctx.device_ctx->cublas_handle(),
                 idbn_blob->shape().elem_cnt(), &alpha,
                 static_cast<const floating_point_type*>(odbn_blob->dptr()), 1,
                 static_cast<floating_point_type*>(idbn_blob->mut_dptr()), 1),
             hipSuccess);
  }
}

INSTANTIATE_GPU_KERNEL_CLASS(CloneKernel);
REGISTER_GPU_KERNEL(OperatorConf::kCloneConf, CloneKernel);

}  // namespace oneflow
