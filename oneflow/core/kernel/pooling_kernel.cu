#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/pooling_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void MaxPoolForward(const int64_t nthreads, const T* in_dptr,
                               T* out_dptr, uint32_t* mask_dptr,
                               const int64_t channels, const int64_t height,
                               const int64_t width, const int64_t pooled_height,
                               const int64_t pooled_width,
                               const int64_t kernel_h, const int64_t kernel_w,
                               const int64_t stride_h, const int64_t stride_w,
                               const int64_t pad_h, const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t pw = index % pooled_width;
    const int64_t ph = (index / pooled_width) % pooled_height;
    const int64_t c = (index / pooled_width / pooled_height) % channels;
    const int64_t n = index / pooled_width / pooled_height / channels;
    int64_t hstart = ph * stride_h - pad_h;
    int64_t wstart = pw * stride_w - pad_w;
    const int64_t hend =
        (hstart + kernel_h < height) ? (hstart + kernel_h) : height;
    const int64_t wend =
        (wstart + kernel_w < width) ? (wstart + kernel_w) : width;
    hstart = (hstart > 0) ? hstart : 0;
    wstart = (wstart > 0) ? wstart : 0;
    const T* const in_slice = in_dptr + (n * channels + c) * height * width;
    T maxval = in_slice[hstart * width + wstart];
    uint32_t maxidx = hstart * width + wstart;
    for (int64_t h = hstart; h < hend; ++h) {
      for (int64_t w = wstart; w < wend; ++w) {
        if (in_slice[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = in_slice[maxidx];
        }
      }
    }
    out_dptr[index] = maxval;
    mask_dptr[index] = maxidx;
  }
}

template<typename T>
__global__ void AvePoolForward(const int64_t nthreads, const T* in_dptr,
                               T* out_dptr, const int64_t channels,
                               const int64_t height, const int64_t width,
                               const int64_t pooled_height,
                               const int64_t pooled_width,
                               const int64_t kernel_h, const int64_t kernel_w,
                               const int64_t stride_h, const int64_t stride_w,
                               const int64_t pad_h, const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t pw = index % pooled_width;
    const int64_t ph = (index / pooled_width) % pooled_height;
    const int64_t c = (index / pooled_width / pooled_height) % channels;
    const int64_t n = index / pooled_width / pooled_height / channels;
    int64_t hstart = ph * stride_h - pad_h;
    int64_t wstart = pw * stride_w - pad_w;
    int64_t hend = (hstart + kernel_h < height + pad_h) ? (hstart + kernel_h)
                                                        : (height + pad_h);
    int64_t wend = (wstart + kernel_w < width + pad_w) ? (wstart + kernel_w)
                                                       : (width + pad_w);
    const int64_t pool_size = (hend - hstart) * (wend - wstart);
    hstart = (hstart > 0) ? hstart : 0;
    wstart = (wstart > 0) ? wstart : 0;
    hend = (hend < height) ? hend : height;
    wend = (wend < width) ? wend : width;
    T aveval = 0;
    const T* const in_slice = in_dptr + (n * channels + c) * height * width;
    for (int64_t h = hstart; h < hend; ++h) {
      for (int64_t w = wstart; w < wend; ++w) {
        aveval += in_slice[h * width + w];
      }
    }
    out_dptr[index] = aveval / pool_size;
  }
}

template<typename T>
__global__ void MaxPoolBackward(const int64_t nthreads, const T* out_diff_dptr,
                                const uint32_t* mask_dptr, T* in_diff_dptr,
                                const int64_t channels, const int64_t height,
                                const int64_t width,
                                const int64_t pooled_height,
                                const int64_t pooled_width,
                                const int64_t kernel_h, const int64_t kernel_w,
                                const int64_t stride_h, const int64_t stride_w,
                                const int64_t pad_h, const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t w = index % width;
    const int64_t h = (index / width) % height;
    const int64_t c = (index / width / height) % channels;
    const int64_t n = index / width / height / channels;
    int64_t phstart =
        (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    int64_t pwstart =
        (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    const int64_t phend = ((h + pad_h) / stride_h + 1 < pooled_height)
                              ? ((h + pad_h) / stride_h + 1)
                              : pooled_height;
    const int64_t pwend = ((w + pad_w) / stride_w + 1 < pooled_width)
                              ? ((w + pad_w) / stride_w + 1)
                              : pooled_width;
    T gradient = 0;
    const int64_t offset = (n * channels + c) * pooled_height * pooled_width;
    const T* const out_diff_slice = out_diff_dptr + offset;
    const uint32_t* const mask_slice = mask_dptr + offset;
    for (int64_t ph = phstart; ph < phend; ++ph) {
      for (int64_t pw = pwstart; pw < pwend; ++pw) {
        if (mask_slice[ph * pooled_width + pw] == h * width + w) {
          gradient += out_diff_slice[ph * pooled_width + pw];
        }
      }
    }
    in_diff_dptr[index] = gradient;
  }
}

template<typename T>
__global__ void AvePoolBackward(const int64_t nthreads, const T* out_diff_dptr,
                                T* in_diff_dptr, const int64_t channels,
                                const int64_t height, const int64_t width,
                                const int64_t pooled_height,
                                const int64_t pooled_width,
                                const int64_t kernel_h, const int64_t kernel_w,
                                const int64_t stride_h, const int64_t stride_w,
                                const int64_t pad_h, const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t w = index % width + pad_w;
    const int64_t h = (index / width) % height + pad_h;
    const int64_t c = (index / width / height) % channels;
    const int64_t n = index / width / height / channels;
    int64_t phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int64_t pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int64_t phend =
        (h / stride_h + 1 < pooled_height) ? (h / stride_h + 1) : pooled_height;
    const int64_t pwend =
        (w / stride_w + 1 < pooled_width) ? (w / stride_w + 1) : pooled_width;
    T gradient = 0;
    const int64_t offset = (n * channels + c) * pooled_height * pooled_width;
    const T* const out_diff_slice = out_diff_dptr + offset;
    for (int64_t ph = phstart; ph < phend; ++ph) {
      for (int64_t pw = pwstart; pw < pwend; ++pw) {
        int64_t hstart = ph * stride_h - pad_h;
        int64_t wstart = pw * stride_w - pad_w;
        int64_t hend = (hstart + kernel_h < height + pad_h)
                           ? (hstart + kernel_h)
                           : (height + pad_h);
        int64_t wend = (wstart + kernel_w < width + pad_w) ? (wstart + kernel_w)
                                                           : (width + pad_w);
        int64_t pool_size = (hend - hstart) * (wend - wstart);
        gradient += out_diff_slice[ph * pooled_width + pw] / pool_size;
      }
    }
    in_diff_dptr[index] = gradient;
  }
}

}  // namespace

template<typename T>
class PoolingKernelUtil<DeviceType::kGPU, T> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PoolingKernelUtil);
  PoolingKernelUtil() = delete;

  static void PoolingForward(const KernelCtx& ctx, const Blob* in_blob,
                             Blob* out_blob, Blob* mask_blob,
                             const PoolingOpConf& pooling_conf) {
    const int64_t count = out_blob->shape().elem_cnt();

    switch (pooling_conf.pool()) {
      case PoolingOpConf::kMax: {
        MaxPoolForward<T>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, in_blob->dptr<T>(), out_blob->mut_dptr<T>(),
                mask_blob->mut_dptr<uint32_t>(), in_blob->shape().At(1),
                in_blob->shape().At(2), in_blob->shape().At(3),
                out_blob->shape().At(2), out_blob->shape().At(3),
                pooling_conf.kernel_size_h(), pooling_conf.kernel_size_w(),
                pooling_conf.stride_h(), pooling_conf.stride_w(),
                pooling_conf.pad_h(), pooling_conf.pad_w());
        break;
      }
      case PoolingOpConf::kAve: {
        AvePoolForward<T>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, in_blob->dptr<T>(), out_blob->mut_dptr<T>(),
                in_blob->shape().At(1), in_blob->shape().At(2),
                in_blob->shape().At(3), out_blob->shape().At(2),
                out_blob->shape().At(3), pooling_conf.kernel_size_h(),
                pooling_conf.kernel_size_w(), pooling_conf.stride_h(),
                pooling_conf.stride_w(), pooling_conf.pad_h(),
                pooling_conf.pad_w());
        break;
      }
      case PoolingOpConf::kStochastic: {
        TODO();
      }
      default: { UNEXPECTED_RUN(); }
    }
  }

  static void PoolingBackward(const KernelCtx& ctx, const Blob* out_diff_blob,
                              const Blob* mask_blob, Blob* in_diff_blob,
                              const PoolingOpConf& pooling_conf) {
    const int64_t count = in_diff_blob->shape().elem_cnt();

    switch (pooling_conf.pool()) {
      case PoolingOpConf::kMax: {
        MaxPoolBackward<T>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, out_diff_blob->dptr<T>(), mask_blob->dptr<uint32_t>(),
                in_diff_blob->mut_dptr<T>(), in_diff_blob->shape().At(1),
                in_diff_blob->shape().At(2), in_diff_blob->shape().At(3),
                out_diff_blob->shape().At(2), out_diff_blob->shape().At(3),
                pooling_conf.kernel_size_h(), pooling_conf.kernel_size_w(),
                pooling_conf.stride_h(), pooling_conf.stride_w(),
                pooling_conf.pad_h(), pooling_conf.pad_w());
        break;
      }
      case PoolingOpConf::kAve: {
        AvePoolBackward<T>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, out_diff_blob->dptr<T>(), in_diff_blob->mut_dptr<T>(),
                in_diff_blob->shape().At(1), in_diff_blob->shape().At(2),
                in_diff_blob->shape().At(3), out_diff_blob->shape().At(2),
                out_diff_blob->shape().At(3), pooling_conf.kernel_size_h(),
                pooling_conf.kernel_size_w(), pooling_conf.stride_h(),
                pooling_conf.stride_w(), pooling_conf.pad_h(),
                pooling_conf.pad_w());
        break;
      }
      case PoolingOpConf::kStochastic: {
        TODO();
      }
      default: { UNEXPECTED_RUN(); }
    }
  }
};

#define INSTANTIATE_POOLING_KERNEL_UTIL(type_cpp, type_proto) \
  template class PoolingKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_POOLING_KERNEL_UTIL, ARITHMETIC_DATA_TYPE_SEQ)

}  // namespace oneflow
