#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/pooling_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void MaxPoolForward(const int64_t nthreads, const T* in_dptr,
                               T* out_dptr, uint32_t* mask_dptr,
                               const int64_t channels, const int64_t height,
                               const int64_t width, const int64_t pooled_height,
                               const int64_t pooled_width,
                               const int64_t kernel_h, const int64_t kernel_w,
                               const int64_t stride_h, const int64_t stride_w,
                               const int64_t pad_h, const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t pw = index % pooled_width;
    const int64_t ph = (index / pooled_width) % pooled_height;
    const int64_t c = (index / pooled_width / pooled_height) % channels;
    const int64_t n = index / pooled_width / pooled_height / channels;
    int64_t hstart = ph * stride_h - pad_h;
    int64_t wstart = pw * stride_w - pad_w;
    const int64_t hend =
        (hstart + kernel_h < height) ? (hstart + kernel_h) : height;
    const int64_t wend =
        (wstart + kernel_w < width) ? (wstart + kernel_w) : width;
    hstart = (hstart > 0) ? hstart : 0;
    wstart = (wstart > 0) ? wstart : 0;
    const T* const in_slice = in_dptr + (n * channels + c) * height * width;
    T maxval = in_slice[hstart * width + wstart];
    uint32_t maxidx = hstart * width + wstart;
    for (int64_t h = hstart; h < hend; ++h) {
      for (int64_t w = wstart; w < wend; ++w) {
        if (in_slice[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = in_slice[maxidx];
        }
      }
    }
    out_dptr[index] = maxval;
    mask_dptr[index] = maxidx;
  }
}

template<typename T>
__global__ void AvePoolForward(const int64_t nthreads, const T* in_dptr,
                               T* out_dptr, const int64_t channels,
                               const int64_t height, const int64_t width,
                               const int64_t pooled_height,
                               const int64_t pooled_width,
                               const int64_t kernel_h, const int64_t kernel_w,
                               const int64_t stride_h, const int64_t stride_w,
                               const int64_t pad_h, const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t pw = index % pooled_width;
    const int64_t ph = (index / pooled_width) % pooled_height;
    const int64_t c = (index / pooled_width / pooled_height) % channels;
    const int64_t n = index / pooled_width / pooled_height / channels;
    int64_t hstart = ph * stride_h - pad_h;
    int64_t wstart = pw * stride_w - pad_w;
    int64_t hend = (hstart + kernel_h < height + pad_h) ? (hstart + kernel_h)
                                                        : (height + pad_h);
    int64_t wend = (wstart + kernel_w < width + pad_w) ? (wstart + kernel_w)
                                                       : (width + pad_w);
    const int64_t pool_size = (hend - hstart) * (wend - wstart);
    hstart = (hstart > 0) ? hstart : 0;
    wstart = (wstart > 0) ? wstart : 0;
    hend = (hend < height) ? hend : height;
    wend = (wend < width) ? wend : width;
    T aveval = 0;
    const T* const in_slice = in_dptr + (n * channels + c) * height * width;
    for (int64_t h = hstart; h < hend; ++h) {
      for (int64_t w = wstart; w < wend; ++w) {
        aveval += in_slice[h * width + w];
      }
    }
    out_dptr[index] = aveval / pool_size;
  }
}

template<typename T>
__global__ void MaxPoolBackward(const int64_t nthreads, const T* out_diff_dptr,
                                const uint32_t* mask_dptr, T* in_diff_dptr,
                                const int64_t channels, const int64_t height,
                                const int64_t width,
                                const int64_t pooled_height,
                                const int64_t pooled_width,
                                const int64_t kernel_h, const int64_t kernel_w,
                                const int64_t stride_h, const int64_t stride_w,
                                const int64_t pad_h, const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t w = index % width;
    const int64_t h = (index / width) % height;
    const int64_t c = (index / width / height) % channels;
    const int64_t n = index / width / height / channels;
    int64_t phstart =
        (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    int64_t pwstart =
        (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    const int64_t phend = ((h + pad_h) / stride_h + 1 < pooled_height)
                              ? ((h + pad_h) / stride_h + 1)
                              : pooled_height;
    const int64_t pwend = ((w + pad_w) / stride_w + 1 < pooled_width)
                              ? ((w + pad_w) / stride_w + 1)
                              : pooled_width;
    T gradient = 0;
    const int64_t offset = (n * channels + c) * pooled_height * pooled_width;
    const T* const out_diff_slice = out_diff_dptr + offset;
    const uint32_t* const mask_slice = mask_dptr + offset;
    for (int64_t ph = phstart; ph < phend; ++ph) {
      for (int64_t pw = pwstart; pw < pwend; ++pw) {
        if (mask_slice[ph * pooled_width + pw] == h * width + w) {
          gradient += out_diff_slice[ph * pooled_width + pw];
        }
      }
    }
    in_diff_dptr[index] = gradient;
  }
}

template<typename T>
__global__ void AvePoolBackward(const int64_t nthreads, const T* out_diff_dptr,
                                T* in_diff_dptr, const int64_t channels,
                                const int64_t height, const int64_t width,
                                const int64_t pooled_height,
                                const int64_t pooled_width,
                                const int64_t kernel_h, const int64_t kernel_w,
                                const int64_t stride_h, const int64_t stride_w,
                                const int64_t pad_h, const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t w = index % width + pad_w;
    const int64_t h = (index / width) % height + pad_h;
    const int64_t c = (index / width / height) % channels;
    const int64_t n = index / width / height / channels;
    int64_t phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int64_t pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int64_t phend =
        (h / stride_h + 1 < pooled_height) ? (h / stride_h + 1) : pooled_height;
    const int64_t pwend =
        (w / stride_w + 1 < pooled_width) ? (w / stride_w + 1) : pooled_width;
    T gradient = 0;
    const int64_t offset = (n * channels + c) * pooled_height * pooled_width;
    const T* const out_diff_slice = out_diff_dptr + offset;
    for (int64_t ph = phstart; ph < phend; ++ph) {
      for (int64_t pw = pwstart; pw < pwend; ++pw) {
        int64_t hstart = ph * stride_h - pad_h;
        int64_t wstart = pw * stride_w - pad_w;
        int64_t hend = (hstart + kernel_h < height + pad_h)
                           ? (hstart + kernel_h)
                           : (height + pad_h);
        int64_t wend = (wstart + kernel_w < width + pad_w) ? (wstart + kernel_w)
                                                           : (width + pad_w);
        int64_t pool_size = (hend - hstart) * (wend - wstart);
        gradient += out_diff_slice[ph * pooled_width + pw] / pool_size;
      }
    }
    in_diff_dptr[index] = gradient;
  }
}

}  // namespace

template<typename T>
class PoolingKernelUtil<DeviceType::kGPU, T> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PoolingKernelUtil);
  PoolingKernelUtil() = delete;

  static void PoolingForward(const KernelCtx& ctx, const Blob* in_blob,
                             Blob* out_blob, Blob* mask_blob,
                             const PoolingOpConf& pooling_conf) {
    const int64_t count = out_blob->shape().elem_cnt();

    switch (pooling_conf.pool()) {
      case PoolingOpConf::kMax: {
        MaxPoolForward<T>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, in_blob->dptr<T>(), out_blob->mut_dptr<T>(),
                mask_blob->mut_dptr<uint32_t>(), in_blob->shape().At(1),
                in_blob->shape().At(2), in_blob->shape().At(3),
                out_blob->shape().At(2), out_blob->shape().At(3),
                pooling_conf.kernel_h(), pooling_conf.kernel_w(),
                pooling_conf.stride_h(), pooling_conf.stride_w(),
                pooling_conf.pad_h(), pooling_conf.pad_w());
        break;
      }
      case PoolingOpConf::kAve: {
        AvePoolForward<T>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, in_blob->dptr<T>(), out_blob->mut_dptr<T>(),
                in_blob->shape().At(1), in_blob->shape().At(2),
                in_blob->shape().At(3), out_blob->shape().At(2),
                out_blob->shape().At(3), pooling_conf.kernel_h(),
                pooling_conf.kernel_w(), pooling_conf.stride_h(),
                pooling_conf.stride_w(), pooling_conf.pad_h(),
                pooling_conf.pad_w());
        break;
      }
      case PoolingOpConf::kStochastic: {
        TODO();
      }
      default: { UNEXPECTED_RUN(); }
    }
  }

  static void PoolingBackward(const KernelCtx& ctx, const Blob* out_diff_blob,
                              const Blob* mask_blob, Blob* in_diff_blob,
                              const PoolingOpConf& pooling_conf) {
    const int64_t count = in_diff_blob->shape().elem_cnt();

    switch (pooling_conf.pool()) {
      case PoolingOpConf::kMax: {
        MaxPoolBackward<T>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, out_diff_blob->dptr<T>(), mask_blob->dptr<uint32_t>(),
                in_diff_blob->mut_dptr<T>(), in_diff_blob->shape().At(1),
                in_diff_blob->shape().At(2), in_diff_blob->shape().At(3),
                out_diff_blob->shape().At(2), out_diff_blob->shape().At(3),
                pooling_conf.kernel_h(), pooling_conf.kernel_w(),
                pooling_conf.stride_h(), pooling_conf.stride_w(),
                pooling_conf.pad_h(), pooling_conf.pad_w());
        break;
      }
      case PoolingOpConf::kAve: {
        AvePoolBackward<T>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, out_diff_blob->dptr<T>(), in_diff_blob->mut_dptr<T>(),
                in_diff_blob->shape().At(1), in_diff_blob->shape().At(2),
                in_diff_blob->shape().At(3), out_diff_blob->shape().At(2),
                out_diff_blob->shape().At(3), pooling_conf.kernel_h(),
                pooling_conf.kernel_w(), pooling_conf.stride_h(),
                pooling_conf.stride_w(), pooling_conf.pad_h(),
                pooling_conf.pad_w());
        break;
      }
      case PoolingOpConf::kStochastic: {
        TODO();
      }
      default: { UNEXPECTED_RUN(); }
    }
  }
};

template<typename T>
CudnnPoolingKernel<DeviceType::kGPU, T>::CudnnPoolingKernel() {
  CudaCheck(hipdnnCreateTensorDescriptor(&in_desc_));
  CudaCheck(hipdnnCreateTensorDescriptor(&out_desc_));
  CudaCheck(hipdnnCreatePoolingDescriptor(&pooling_desc_));
}

template<typename T>
CudnnPoolingKernel<DeviceType::kGPU, T>::~CudnnPoolingKernel() {
  CudaCheck(hipdnnDestroyTensorDescriptor(in_desc_));
  CudaCheck(hipdnnDestroyTensorDescriptor(out_desc_));
  CudaCheck(hipdnnDestroyPoolingDescriptor(pooling_desc_));
}

template<typename T>
void CudnnPoolingKernel<DeviceType::kGPU, T>::InitFromOpProto(
    const OperatorProto& op_proto) {
  Kernel::InitFromOpProto(op_proto);

  const auto pooling_conf = op()->op_conf().pooling_conf();

  switch (pooling_conf.pool()) {
    case PoolingOpConf::kMax: {
      pooling_mode_ = HIPDNN_POOLING_MAX;
      break;
    }
    case PoolingOpConf::kAve: {
      pooling_mode_ = HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
      break;
    }
    default: { UNEXPECTED_RUN(); }
  }

  CudaCheck(hipdnnSetPooling2dDescriptor(
      pooling_desc_, pooling_mode_, HIPDNN_PROPAGATE_NAN,
      pooling_conf.kernel_h(), pooling_conf.kernel_w(), pooling_conf.pad_h(),
      pooling_conf.pad_w(), pooling_conf.stride_h(), pooling_conf.stride_w()));
}

template<typename T>
void CudnnPoolingKernel<DeviceType::kGPU, T>::Forward(
    const KernelCtx& ctx,
    std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  const PoolingOpConf& pooling_conf = op()->op_conf().pooling_conf();

  const Blob* in_blob = BnInOp2Blob("in");
  Blob* out_blob = BnInOp2Blob("out");
  CopyDataIdFromSoleIbToAllObIfNeed<DeviceType::kGPU>(ctx, BnInOp2Blob);

  CudaCheck(hipdnnSetTensor4dDescriptor(
      in_desc_, HIPDNN_TENSOR_NCHW, cudnn::DataType<T>::type,
      in_blob->shape().At(0), in_blob->shape().At(1), in_blob->shape().At(2),
      in_blob->shape().At(3)));
  CudaCheck(hipdnnSetTensor4dDescriptor(
      out_desc_, HIPDNN_TENSOR_NCHW, cudnn::DataType<T>::type,
      out_blob->shape().At(0), out_blob->shape().At(1), out_blob->shape().At(2),
      out_blob->shape().At(3)));

  CudaCheck(hipdnnPoolingForward(ctx.device_ctx->cudnn_handle(), pooling_desc_,
                                cudnn::DataType<T>::one, in_desc_,
                                in_blob->dptr<T>(), cudnn::DataType<T>::zero,
                                out_desc_, out_blob->mut_dptr<T>()));
}

template<typename T>
void CudnnPoolingKernel<DeviceType::kGPU, T>::Backward(
    const KernelCtx& ctx,
    std::function<Blob*(const std::string&)> BnInOp2Blob) const {
  const PoolingOpConf& pooling_conf = op()->op_conf().pooling_conf();
  const Blob* out_diff_blob = BnInOp2Blob("out_diff");
  const Blob* in_blob = BnInOp2Blob("in");
  const Blob* out_blob = BnInOp2Blob("out");
  Blob* in_diff_blob = BnInOp2Blob("in_diff");
  if (in_diff_blob == nullptr) { return; }
  Memset<DeviceType::kGPU>(ctx.device_ctx, in_diff_blob->mut_dptr(), 0,
                           in_diff_blob->ByteSizeOfDataField());

  CudaCheck(hipdnnPoolingBackward(
      ctx.device_ctx->cudnn_handle(), pooling_desc_, cudnn::DataType<T>::one,
      out_desc_, out_blob->dptr<T>(), out_desc_, out_diff_blob->dptr<T>(),
      in_desc_, in_blob->dptr<T>(), cudnn::DataType<T>::zero, in_desc_,
      in_diff_blob->mut_dptr<T>()));
}

#ifdef USE_CUDNN
#define INSTANTIATE_POOLING_KERNEL(type_cpp, type_proto) \
  template class CudnnPoolingKernel<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_POOLING_KERNEL, FLOATING_DATA_TYPE_SEQ)
#endif
#define INSTANTIATE_POOLING_KERNEL_UTIL(type_cpp, type_proto) \
  template class PoolingKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_POOLING_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
