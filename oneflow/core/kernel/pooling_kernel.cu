#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/pooling_kernel.h"

namespace oneflow {

namespace {

template<typename FloatingPointType>
__global__ void MaxPoolForward(const int64_t nthreads,
                               const FloatingPointType* in_dptr,
                               FloatingPointType* out_dptr, uint32_t* mask_dptr,
                               const int64_t channels, const int64_t height,
                               const int64_t width, const int64_t pooled_height,
                               const int64_t pooled_width,
                               const int64_t kernel_h, const int64_t kernel_w,
                               const int64_t stride_h, const int64_t stride_w,
                               const int64_t pad_h, const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t pw = index % pooled_width;
    const int64_t ph = (index / pooled_width) % pooled_height;
    const int64_t c = (index / pooled_width / pooled_height) % channels;
    const int64_t n = index / pooled_width / pooled_height / channels;
    int64_t hstart = ph * stride_h - pad_h;
    int64_t wstart = pw * stride_w - pad_w;
    const int64_t hend =
        (hstart + kernel_h < height) ? (hstart + kernel_h) : height;
    const int64_t wend =
        (wstart + kernel_w < width) ? (wstart + kernel_w) : width;
    hstart = (hstart > 0) ? hstart : 0;
    wstart = (wstart > 0) ? wstart : 0;
    const FloatingPointType* const in_slice =
        in_dptr + (n * channels + c) * height * width;
    FloatingPointType maxval = in_slice[hstart * width + wstart];
    uint32_t maxidx = hstart * width + wstart;
    for (int64_t h = hstart; h < hend; ++h) {
      for (int64_t w = wstart; w < wend; ++w) {
        if (in_slice[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = in_slice[maxidx];
        }
      }
    }
    out_dptr[index] = maxval;
    mask_dptr[index] = maxidx;
  }
}

template<typename FloatingPointType>
__global__ void AvePoolForward(const int64_t nthreads,
                               const FloatingPointType* in_dptr,
                               FloatingPointType* out_dptr,
                               const int64_t channels, const int64_t height,
                               const int64_t width, const int64_t pooled_height,
                               const int64_t pooled_width,
                               const int64_t kernel_h, const int64_t kernel_w,
                               const int64_t stride_h, const int64_t stride_w,
                               const int64_t pad_h, const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t pw = index % pooled_width;
    const int64_t ph = (index / pooled_width) % pooled_height;
    const int64_t c = (index / pooled_width / pooled_height) % channels;
    const int64_t n = index / pooled_width / pooled_height / channels;
    int64_t hstart = ph * stride_h - pad_h;
    int64_t wstart = pw * stride_w - pad_w;
    int64_t hend = (hstart + kernel_h < height + pad_h) ? (hstart + kernel_h)
                                                        : (height + pad_h);
    int64_t wend = (wstart + kernel_w < width + pad_w) ? (wstart + kernel_w)
                                                       : (width + pad_w);
    const int64_t pool_size = (hend - hstart) * (wend - wstart);
    hstart = (hstart > 0) ? hstart : 0;
    wstart = (wstart > 0) ? wstart : 0;
    hend = (hend < height) ? hend : height;
    wend = (wend < width) ? wend : width;
    FloatingPointType aveval = 0;
    const FloatingPointType* const in_slice =
        in_dptr + (n * channels + c) * height * width;
    for (int64_t h = hstart; h < hend; ++h) {
      for (int64_t w = wstart; w < wend; ++w) {
        aveval += in_slice[h * width + w];
      }
    }
    out_dptr[index] = aveval / pool_size;
  }
}

template<typename FloatingPointType>
__global__ void MaxPoolBackward(
    const int64_t nthreads, const FloatingPointType* out_diff_dptr,
    const uint32_t* mask_dptr, FloatingPointType* in_diff_dptr,
    const int64_t channels, const int64_t height, const int64_t width,
    const int64_t pooled_height, const int64_t pooled_width,
    const int64_t kernel_h, const int64_t kernel_w, const int64_t stride_h,
    const int64_t stride_w, const int64_t pad_h, const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t w = index % width;
    const int64_t h = (index / width) % height;
    const int64_t c = (index / width / height) % channels;
    const int64_t n = index / width / height / channels;
    int64_t phstart =
        (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    int64_t pwstart =
        (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    const int64_t phend = ((h + pad_h) / stride_h + 1 < pooled_height)
                              ? ((h + pad_h) / stride_h + 1)
                              : pooled_height;
    const int64_t pwend = ((w + pad_w) / stride_w + 1 < pooled_width)
                              ? ((w + pad_w) / stride_w + 1)
                              : pooled_width;
    FloatingPointType gradient = 0;
    const int64_t offset = (n * channels + c) * pooled_height * pooled_width;
    const FloatingPointType* const out_diff_slice = out_diff_dptr + offset;
    const uint32_t* const mask_slice = mask_dptr + offset;
    for (int64_t ph = phstart; ph < phend; ++ph) {
      for (int64_t pw = pwstart; pw < pwend; ++pw) {
        if (mask_slice[ph * pooled_width + pw] == h * width + w) {
          gradient += out_diff_slice[ph * pooled_width + pw];
        }
      }
    }
    in_diff_dptr[index] = gradient;
  }
}

template<typename FloatingPointType>
__global__ void AvePoolBackward(
    const int64_t nthreads, const FloatingPointType* out_diff_dptr,
    FloatingPointType* in_diff_dptr, const int64_t channels,
    const int64_t height, const int64_t width, const int64_t pooled_height,
    const int64_t pooled_width, const int64_t kernel_h, const int64_t kernel_w,
    const int64_t stride_h, const int64_t stride_w, const int64_t pad_h,
    const int64_t pad_w) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t w = index % width + pad_w;
    const int64_t h = (index / width) % height + pad_h;
    const int64_t c = (index / width / height) % channels;
    const int64_t n = index / width / height / channels;
    int64_t phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int64_t pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int64_t phend =
        (h / stride_h + 1 < pooled_height) ? (h / stride_h + 1) : pooled_height;
    const int64_t pwend =
        (w / stride_w + 1 < pooled_width) ? (w / stride_w + 1) : pooled_width;
    FloatingPointType gradient = 0;
    const int64_t offset = (n * channels + c) * pooled_height * pooled_width;
    const FloatingPointType* const out_diff_slice = out_diff_dptr + offset;
    for (int64_t ph = phstart; ph < phend; ++ph) {
      for (int64_t pw = pwstart; pw < pwend; ++pw) {
        int64_t hstart = ph * stride_h - pad_h;
        int64_t wstart = pw * stride_w - pad_w;
        int64_t hend = (hstart + kernel_h < height + pad_h)
                           ? (hstart + kernel_h)
                           : (height + pad_h);
        int64_t wend = (wstart + kernel_w < width + pad_w) ? (wstart + kernel_w)
                                                           : (width + pad_w);
        int64_t pool_size = (hend - hstart) * (wend - wstart);
        gradient += out_diff_slice[ph * pooled_width + pw] / pool_size;
      }
    }
    in_diff_dptr[index] = gradient;
  }
}

}  // namespace

template<typename FloatingPointType>
class PoolingKernelUtil<DeviceType::kGPU, FloatingPointType> final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PoolingKernelUtil);
  PoolingKernelUtil() = delete;

  static void PoolingForward(const KernelCtx& ctx, const Blob* in_blob,
                             Blob* out_blob, Blob* mask_blob,
                             const PoolingOpConf& pooling_conf) {
    const int64_t count = out_blob->shape().elem_cnt();

    switch (pooling_conf.pool()) {
      case PoolingOpConf::MAX: {
        MaxPoolForward<FloatingPointType>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, in_blob->dptr<FloatingPointType>(),
                out_blob->mut_dptr<FloatingPointType>(),
                mask_blob->mut_dptr<uint32_t>(), in_blob->shape().At(1),
                in_blob->shape().At(2), in_blob->shape().At(3),
                out_blob->shape().At(2), out_blob->shape().At(3),
                pooling_conf.kernel_size(0), pooling_conf.kernel_size(1),
                pooling_conf.stride(0), pooling_conf.stride(1),
                pooling_conf.pad(0), pooling_conf.pad(1));
        break;
      }
      case PoolingOpConf::AVE: {
        AvePoolForward<FloatingPointType>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, in_blob->dptr<FloatingPointType>(),
                out_blob->mut_dptr<FloatingPointType>(), in_blob->shape().At(1),
                in_blob->shape().At(2), in_blob->shape().At(3),
                out_blob->shape().At(2), out_blob->shape().At(3),
                pooling_conf.kernel_size(0), pooling_conf.kernel_size(1),
                pooling_conf.stride(0), pooling_conf.stride(1),
                pooling_conf.pad(0), pooling_conf.pad(1));
        break;
      }
      case PoolingOpConf::STOCHASTIC: {
        TODO();
      }
      default: { UNEXPECTED_RUN(); }
    }
  }

  static void PoolingBackward(const KernelCtx& ctx, const Blob* out_diff_blob,
                              const Blob* mask_blob, Blob* in_diff_blob,
                              const PoolingOpConf& pooling_conf) {
    const int64_t count = in_diff_blob->shape().elem_cnt();

    switch (pooling_conf.pool()) {
      case PoolingOpConf::MAX: {
        MaxPoolBackward<FloatingPointType>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, out_diff_blob->dptr<FloatingPointType>(),
                mask_blob->dptr<uint32_t>(),
                in_diff_blob->mut_dptr<FloatingPointType>(),
                in_diff_blob->shape().At(1), in_diff_blob->shape().At(2),
                in_diff_blob->shape().At(3), out_diff_blob->shape().At(2),
                out_diff_blob->shape().At(3), pooling_conf.kernel_size(0),
                pooling_conf.kernel_size(1), pooling_conf.stride(0),
                pooling_conf.stride(1), pooling_conf.pad(0),
                pooling_conf.pad(1));
        break;
      }
      case PoolingOpConf::AVE: {
        AvePoolBackward<FloatingPointType>
            <<<BlocksNum4ThreadsNum(count), kCudaThreadsNumPerBlock, 0,
               ctx.device_ctx->hip_stream()>>>(
                count, out_diff_blob->dptr<FloatingPointType>(),
                in_diff_blob->mut_dptr<FloatingPointType>(),
                in_diff_blob->shape().At(1), in_diff_blob->shape().At(2),
                in_diff_blob->shape().At(3), out_diff_blob->shape().At(2),
                out_diff_blob->shape().At(3), pooling_conf.kernel_size(0),
                pooling_conf.kernel_size(1), pooling_conf.stride(0),
                pooling_conf.stride(1), pooling_conf.pad(0),
                pooling_conf.pad(1));
        break;
      }
      case PoolingOpConf::STOCHASTIC: {
        TODO();
      }
      default: { UNEXPECTED_RUN(); }
    }
  }
};

INSTANTIATE_GPU_KERNEL_UTIL_CLASS(PoolingKernelUtil);

}  // namespace oneflow
