#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/pooling_kernel.h"

namespace oneflow {

template<typename T>
void PoolingKernel<DeviceType::kGPU, T>::PoolingForward(const KernelCtx& kernel_ctx,
                                                        const PoolingCtx& pooling_ctx,
                                                        const Blob* in_blob, Blob* out_blob) const {
  CudaCheck(hipdnnPoolingForward(
      kernel_ctx.device_ctx->cudnn_handle(), pooling_ctx.cudnn_pooling_desc(), CudnnSPOnePtr<T>(),
      pooling_ctx.cudnn_in_tensor_desc(), in_blob->dptr(), CudnnSPZeroPtr<T>(),
      pooling_ctx.cudnn_out_tensor_desc(), out_blob->mut_dptr()));
}

template<typename T>
void PoolingKernel<DeviceType::kGPU, T>::PoolingBackward(const KernelCtx& kernel_ctx,
                                                         const PoolingCtx& pooling_ctx,
                                                         const Blob* out_diff_blob,
                                                         const Blob* out_blob, const Blob* in_blob,
                                                         Blob* in_diff_blob) const {
  CudaCheck(hipdnnPoolingBackward(
      kernel_ctx.device_ctx->cudnn_handle(), pooling_ctx.cudnn_pooling_desc(), CudnnSPOnePtr<T>(),
      pooling_ctx.cudnn_out_tensor_desc(), out_blob->dptr(), pooling_ctx.cudnn_out_tensor_desc(),
      out_diff_blob->dptr(), pooling_ctx.cudnn_in_tensor_desc(), in_blob->dptr(),
      CudnnSPZeroPtr<T>(), pooling_ctx.cudnn_in_tensor_desc(), in_diff_blob->mut_dptr()));
}

#define INSTANTIATE_POOLING_KERNEL(type_cpp, type_proto) \
  template class PoolingKernel<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_POOLING_KERNEL, FLOATING_DATA_TYPE_SEQ FLOAT16_DATA_TYPE_SEQ)

}  // namespace oneflow
