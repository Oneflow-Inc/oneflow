#include "oneflow/core/kernel/conv_filter_grad_kernel.h"
#include "oneflow/core/operator/conv_op.h"
#include "oneflow/core/device/cudnn_conv_util.h"

namespace oneflow {

template<typename T>
struct ConvFilterGradKernelUtil<DeviceType::kGPU, T> final {
  static void Compute(DeviceCtx *ctx, const ConvFilterGradKernelConf &kernel_conf,
                      const ConvConf &conf, const Blob *x, const Blob *dy, Blob *filter_diff,
                      Blob *buf, bool deterministic, bool heuristic) {
    CudnnConvArgs args(conf, ctx->cudnn_handle(), x, dy, filter_diff, buf, deterministic,
                       heuristic);
    hipdnnConvolutionBwdFilterAlgo_t algo;
    size_t work_space_size = 0;
    if (kernel_conf.has_cudnn_bwd_filter_algo()) {
      algo = static_cast<hipdnnConvolutionBwdFilterAlgo_t>(kernel_conf.cudnn_bwd_filter_algo());
      work_space_size = args.ws_size;
    } else {
      auto algo_perf = FindCudnnConvAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>(args);
      algo = algo_perf->algo;
      work_space_size = algo_perf->memory;
    }
    CudaCheck(hipdnnConvolutionBackwardFilter(
        args.handle, CudnnSPOnePtr<T>(), args.xdesc.Get(), args.x_dptr, args.ydesc.Get(),
        args.y_dptr, args.cdesc.Get(), algo, args.work_space, work_space_size, CudnnSPZeroPtr<T>(),
        args.wdesc.Get(), args.w_dptr));
  }
};

#define INSTANTIATE_CONV_FILTER_GRAD_KERNEL_UTIL(type_cpp, type_proto) \
  template struct ConvFilterGradKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_CONV_FILTER_GRAD_KERNEL_UTIL,
                     FLOATING_DATA_TYPE_SEQ FLOAT16_DATA_TYPE_SEQ)

}  // namespace oneflow
