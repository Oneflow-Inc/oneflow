#include "oneflow/core/kernel/conv_filter_grad_kernel.h"
#include "oneflow/core/operator/conv_op.h"

namespace oneflow {

template<typename T>
struct ConvFilterGradKernelUtil<DeviceType::kGPU, T> final {
  static void Compute(DeviceCtx *ctx, const ConvFilterGradKernelConf &kernel_conf,
                      const ConvConf &conf, const Blob *x, const Blob *dy, Blob *filter_diff,
                      Blob *buf) {
    CudnnTensorDesc x_desc(x->data_type(), x->shape(), conf.data_format());
    CudnnTensorDesc dy_desc(dy->data_type(), dy->shape(), conf.data_format());
    CudnnFilterDesc filter_diff_desc(filter_diff->data_type(), filter_diff->shape(),
                                     conf.data_format());
    CudnnConvDesc conv_desc(GetConvDescDataType(x->data_type()), x->shape(), conf);
    CudaCheck(hipdnnConvolutionBackwardFilter(
        ctx->cudnn_handle(), CudnnSPOnePtr<T>(), x_desc.Get(), x->dptr<T>(), dy_desc.Get(),
        dy->dptr<T>(), conv_desc.Get(),
        static_cast<hipdnnConvolutionBwdFilterAlgo_t>(kernel_conf.cudnn_bwd_filter_algo()),
        buf->mut_dptr(), buf->ByteSizeOfDataContentField(), CudnnSPZeroPtr<T>(),
        filter_diff_desc.Get(), filter_diff->mut_dptr<T>()));
  }
};

#define INSTANTIATE_CONV_FILTER_GRAD_KERNEL_UTIL(type_cpp, type_proto) \
  template struct ConvFilterGradKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_CONV_FILTER_GRAD_KERNEL_UTIL,
                     FLOATING_DATA_TYPE_SEQ FLOAT16_DATA_TYPE_SEQ)

}  // namespace oneflow
