#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/bias_add_kernel.h"

namespace oneflow {

namespace {

template<typename T, typename Index>
__global__ void BiasAddGpu(const Index elem_cnt, const Index bias_size, const Index inner_size,
                           const T* x, const T* bias, T* y) {
  const Index block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) { y[i] = x[i] + bias[(i % block_size) / inner_size]; }
}

template<typename Index>
__global__ void BiasAddForwardGpuHalf(const Index elem_cnt, const Index bias_size,
                                      const Index inner_size, const half* x, const half* bias,
                                      half* y) {
  const Index block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    y[i] = __hadd(x[i], bias[(i % block_size) / inner_size]);
  }
}

template<typename T, typename Index>
__global__ void InplaceBiasAddGpu(const Index elem_cnt, const Index bias_size,
                                  const Index inner_size, const T* bias, T* y) {
  const Index block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) { y[i] += bias[(i % block_size) / inner_size]; }
}

template<typename T, typename Index>
struct BiasAddGpuHelper final {
  static void BiasAdd(DeviceCtx* ctx, const Index elem_cnt, const Index bias_size,
                      const Index inner_size, const T* x, const T* bias, T* y) {
    if (x == y) {
      InplaceBiasAddGpu<T, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              elem_cnt, bias_size, inner_size, bias, y);
    } else {
      BiasAddGpu<T, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              elem_cnt, bias_size, inner_size, x, bias, y);
    }
  }
};

}  // namespace

template<typename T>
struct BiasAddUtil<DeviceType::kGPU, T> {
  static void BiasAdd(DeviceCtx* ctx, int64_t outer_size, int64_t bias_size, int64_t inner_size,
                      const T* x, const T* bias, T* y) {
    const int64_t elem_cnt = outer_size * bias_size * inner_size;
    if (IsKernelSafeInt32(elem_cnt)) {
      BiasAddGpuHelper<T, int32_t>::BiasAdd(ctx, elem_cnt, bias_size, inner_size, x, bias, y);
    } else {
      BiasAddGpuHelper<T, int64_t>::BiasAdd(ctx, elem_cnt, bias_size, inner_size, x, bias, y);
    }
  }
};

class BiasAddGpuHalfKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(BiasAddGpuHalfKernel);
  BiasAddGpuHalfKernel() = default;
  ~BiasAddGpuHalfKernel() = default;

 private:
  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    const Blob* a_blob = BnInOp2Blob("a");
    const Blob* b_blob = BnInOp2Blob("b");
    Blob* out_blob = BnInOp2Blob("out");
    const BiasAddOpConf& conf = this->op_conf().bias_add_conf();
    const int32_t bias_add_axis = conf.axis();
    const int64_t outer_size = a_blob->shape().Count(0, bias_add_axis);
    const int64_t bias_size = a_blob->shape().At(bias_add_axis);
    const int64_t inner_size = a_blob->shape().Count(bias_add_axis + 1);
    const int64_t elem_cnt = outer_size * bias_size * inner_size;
    if (IsKernelSafeInt32(elem_cnt)) {
      BiasAddForwardGpuHalf<int32_t><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                       ctx.device_ctx->hip_stream()>>>(
          elem_cnt, bias_size, inner_size, reinterpret_cast<const half*>(a_blob->dptr<float16>()),
          reinterpret_cast<const half*>(b_blob->dptr<float16>()),
          reinterpret_cast<half*>(out_blob->mut_dptr<float16>()));

    } else {
      BiasAddForwardGpuHalf<int64_t><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                       ctx.device_ctx->hip_stream()>>>(
          elem_cnt, bias_size, inner_size, reinterpret_cast<const half*>(a_blob->dptr<float16>()),
          reinterpret_cast<const half*>(b_blob->dptr<float16>()),
          reinterpret_cast<half*>(out_blob->mut_dptr<float16>()));
    }
  }

  const PbMessage& GetCustomizedOpConf() const override { return this->op_conf().bias_add_conf(); }
};

REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kBiasAddConf, DeviceType::kGPU, float16,
                                      BiasAddGpuHalfKernel)

#define INITIATE_BIAS_ADD_KERNEL_UTIL_GPU_IMPL(type_cpp, type_proto) \
  template struct BiasAddUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INITIATE_BIAS_ADD_KERNEL_UTIL_GPU_IMPL, ARITHMETIC_DATA_TYPE_SEQ);
#undef INITIATE_BIAS_ADD_KERNEL_UTIL_GPU_IMPL

}  // namespace oneflow
