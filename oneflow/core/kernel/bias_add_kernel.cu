#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/bias_add_kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void BiasAddGpu(const int64_t elem_cnt, const int64_t bias_size,
                           const int64_t inner_size, const T* x, const T* bias, T* y) {
  const int64_t block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) { y[i] = x[i] + bias[(i % block_size) / inner_size]; }
}

__global__ void BiasAddForwardGpuHalf(const int64_t elem_cnt, const int64_t bias_size,
                                      const int64_t inner_size, const half* x, const half* bias,
                                      half* y) {
  const int64_t block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) { y[i] = __hadd(x[i], bias[(i % block_size) / inner_size]); }
}

}  // namespace

template<typename T>
struct BiasAddUtil<DeviceType::kGPU, T> {
  static void BiasAdd(DeviceCtx* ctx, int64_t outer_size, int64_t bias_size, int64_t inner_size,
                      const T* x, const T* bias, T* y) {
    const int64_t elem_cnt = outer_size * bias_size * inner_size;
    BiasAddGpu<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
        elem_cnt, bias_size, inner_size, x, bias, y);
  }
};

class BiasAddGpuHalfKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(BiasAddGpuHalfKernel);
  BiasAddGpuHalfKernel() = default;
  ~BiasAddGpuHalfKernel() = default;

 private:
  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    const Blob* a_blob = BnInOp2Blob("a");
    const Blob* b_blob = BnInOp2Blob("b");
    Blob* out_blob = BnInOp2Blob("out");
    const BiasAddOpConf& conf = this->op_conf().bias_add_conf();
    const int32_t bias_add_axis = conf.axis();
    const int64_t outer_size = a_blob->shape().Count(0, bias_add_axis);
    const int64_t bias_size = a_blob->shape().At(bias_add_axis);
    const int64_t inner_size = a_blob->shape().Count(bias_add_axis + 1);
    const int64_t elem_cnt = outer_size * bias_size * inner_size;
    BiasAddForwardGpuHalf<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                            ctx.device_ctx->hip_stream()>>>(
        elem_cnt, bias_size, inner_size, reinterpret_cast<const half*>(a_blob->dptr<float16>()),
        reinterpret_cast<const half*>(b_blob->dptr<float16>()),
        reinterpret_cast<half*>(out_blob->mut_dptr<float16>()));
  }

  const PbMessage& GetCustomizedOpConf() const override { return this->op_conf().bias_add_conf(); }
};

REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kBiasAddConf, DeviceType::kGPU, float16,
                                      BiasAddGpuHalfKernel)

#define INITIATE_BIAS_ADD_KERNEL_UTIL_GPU_IMPL(type_cpp, type_proto) \
  template struct BiasAddUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INITIATE_BIAS_ADD_KERNEL_UTIL_GPU_IMPL, ARITHMETIC_DATA_TYPE_SEQ);
#undef INITIATE_BIAS_ADD_KERNEL_UTIL_GPU_IMPL

}  // namespace oneflow
