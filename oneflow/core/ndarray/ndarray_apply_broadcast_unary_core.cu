#include "hip/hip_runtime.h"
#include "oneflow/core/ndarray/ndarray_apply_broadcast_unary_core.h"

namespace oneflow {

namespace {

template<typename T, int NDIMS, const T (*unary_func)(const T)>
__global__ void GpuBroadcastUnaryFunc(const XpuVarNdarray<T> y, const XpuVarNdarray<const T> x) {
  NdArrayApplyBroadcastUnaryCore<T, NDIMS, unary_func>::Apply(y, x);
}

}  // namespace

template<typename T, int NDIMS, const T (*unary_func)(const T)>
struct NdArrayApplyBroadcastUnaryCoreWrapper<DeviceType::kGPU, T, NDIMS, unary_func> final {
  static void Apply(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    size_t n = y.host_shape().HostElemNum();
    RUN_CUDA_KERNEL((GpuBroadcastUnaryFunc<T, NDIMS, unary_func>), ctx, n, y, x);
  }
};

#define INSTANTIATE_BROADCAST_UNARY_FUNC(dtype_pair, NDIMS, unary_func) \
  template struct NdArrayApplyBroadcastUnaryCoreWrapper<                \
      DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype_pair), NDIMS, unary_func>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_BROADCAST_UNARY_FUNC, ARITHMETIC_DATA_TYPE_SEQ,
                                 DIM_SEQ, ARITHMETIC_UNARY_FUNC_SEQ)
}  // namespace oneflow
