#include "hip/hip_runtime.h"
#include "oneflow/core/ndarray/ndarray_apply_unary_core.h"
#include "oneflow/core/ndarray/unary_func.h"

namespace oneflow {

namespace {

template<typename T, const T (*unary_func)(const T)>
__global__ void NdArrayApplyUnaryImplaceApplyGpu(T* ptr, size_t n) {
  NdArrayApplyUnaryCore<T, unary_func>::ImplaceApply(ptr, n);
}

}  // namespace

template<typename T, const T (*unary_func)(const T)>
struct NdArrayApplyUnaryCoreWrapper<DeviceType::kGPU, T, unary_func> final {
  static void ImplaceApply(DeviceCtx* ctx, const XpuVarNdarray<T>& y) {
    size_t n = y.host_shape().HostElemNum();
    RUN_CUDA_KERNEL((NdArrayApplyUnaryImplaceApplyGpu<T, unary_func>), ctx, n, y.host_ptr(), n);
  }
};

#define INSTANTIATE_NDARRAY_APPLY_UNARY_CORE(dtype_pair, unary_func)                           \
  template struct NdArrayApplyUnaryCoreWrapper<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype_pair), \
                                               unary_func>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_NDARRAY_APPLY_UNARY_CORE, ARITHMETIC_DATA_TYPE_SEQ,
                                 ARITHMETIC_UNARY_FUNC_SEQ)

}  // namespace oneflow
