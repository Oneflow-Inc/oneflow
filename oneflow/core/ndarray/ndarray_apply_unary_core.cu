#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ndarray/ndarray_apply_unary_core.h"
#include "oneflow/core/ndarray/unary_func.h"

namespace oneflow {

namespace {

template<typename T, template<typename> class unary_func>
__global__ void NdarrayApplyUnaryInplaceApplyGpu(T* ptr, size_t n) {
  NdarrayApplyUnaryCore<T, unary_func>::InplaceApply(ptr, n);
}

}  // namespace

template<typename T, template<typename> class unary_func>
struct NdarrayApplyUnaryCoreWrapper<DeviceType::kGPU, T, unary_func> final {
  static void InplaceApply(DeviceCtx* ctx, const XpuVarNdarray<T>& y) {
    size_t n = y.host_shape().HostElemNum();
    RUN_CUDA_KERNEL((NdarrayApplyUnaryInplaceApplyGpu<T, unary_func>), ctx, n, y.host_ptr(), n);
  }
};

#define INSTANTIATE_NDARRAY_APPLY_UNARY_CORE(dtype_pair, unary_func)                           \
  template struct NdarrayApplyUnaryCoreWrapper<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype_pair), \
                                               unary_func>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_NDARRAY_APPLY_UNARY_CORE,
                                 ARITHMETIC_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ,
                                 ARITHMETIC_UNARY_FUNC_SEQ);

}  // namespace oneflow
