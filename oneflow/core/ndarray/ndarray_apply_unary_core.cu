#include "hip/hip_runtime.h"
#include "oneflow/core/ndarray/ndarray_apply_unary_core.h"
#include "oneflow/core/ndarray/unary_func.h"

namespace oneflow {

namespace {

template<typename T, const T (*unary_func)(const T)>
__global__ void NdarrayApplyUnaryImplaceApplyGpu(T* ptr, size_t n) {
  NdarrayApplyUnaryCore<T, unary_func>::ImplaceApply(ptr, n);
}

}  // namespace

template<typename T, const T (*unary_func)(const T)>
struct NdarrayApplyUnaryCoreWrapper<DeviceType::kGPU, T, unary_func> final {
  static void ImplaceApply(DeviceCtx* ctx, const XpuVarNdarray<T>& y) {
    size_t n = y.host_shape().HostElemNum();
    RUN_CUDA_KERNEL((NdarrayApplyUnaryImplaceApplyGpu<T, unary_func>), ctx, n, y.host_ptr(), n);
  }
};

#define INSTANTIATE_NDARRAY_APPLY_UNARY_CORE(dtype_pair, unary_func)                           \
  template struct NdarrayApplyUnaryCoreWrapper<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype_pair), \
                                               unary_func>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_NDARRAY_APPLY_UNARY_CORE, ARITHMETIC_DATA_TYPE_SEQ,
                                 ARITHMETIC_UNARY_FUNC_SEQ)

}  // namespace oneflow
