#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ndarray/ndarray_apply_binary_core.h"
#include "oneflow/core/ndarray/binary_func.h"

namespace oneflow {

namespace {

template<typename T, template<typename> class binary_func>
__global__ void NdarrayApplyBinaryApplyGpu(size_t n,
                                           typename BinaryFuncTrait<binary_func, T>::return_type* y,
                                           const T* a, const T* b) {
  NdarrayApplyBinaryCore<T, binary_func>::Apply(n, y, a, b);
}

template<typename T, template<typename> class binary_func>
__global__ void NdarrayApplyBinaryInplaceApplyGpu(size_t n, T* y, const T* x) {
  NdarrayApplyBinaryCore<T, binary_func>::InplaceApply(n, y, x);
}

}  // namespace

template<typename T, template<typename> class binary_func>
struct NdarrayApplyBinaryCoreWrapper<DeviceType::kGPU, T, binary_func> final {
  static void Apply(DeviceCtx* ctx,
                    const XpuVarNdarray<typename BinaryFuncTrait<binary_func, T>::return_type>& y,
                    const XpuVarNdarray<const T>& a, const XpuVarNdarray<const T>& b) {
    size_t n = y.host_shape().HostElemNum();
    RUN_CUDA_KERNEL((NdarrayApplyBinaryApplyGpu<T, binary_func>), ctx, n, n, y.host_ptr(),
                    a.host_ptr(), b.host_ptr());
  }
  static void InplaceApply(DeviceCtx* ctx, const XpuVarNdarray<T>& y,
                           const XpuVarNdarray<const T>& x) {
    size_t n = y.host_shape().HostElemNum();
    RUN_CUDA_KERNEL((NdarrayApplyBinaryInplaceApplyGpu<T, binary_func>), ctx, n, n, y.host_ptr(),
                    x.host_ptr());
  }
};

#define INSTANTIATE_NDARRAY_APPLY_BINARY_CORE(dtype_pair, binary_func)                          \
  template struct NdarrayApplyBinaryCoreWrapper<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype_pair), \
                                                binary_func>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_NDARRAY_APPLY_BINARY_CORE,
                                 ARITHMETIC_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ, BINARY_FUNC_SEQ);

}  // namespace oneflow
