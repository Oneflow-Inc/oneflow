#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/ndarray/ndarray_reduce_impl.h"
#include "oneflow/core/ndarray/binary_func.h"
#include "oneflow/core/common/preprocessor.h"
#include "oneflow/core/common/shape.h"

namespace oneflow {

namespace {

template<typename T, const T (*binary_func)(const T, const T)>
void __global__ NdarrayMatrixColReduceNaiveCudaKernel(T* y_ptr, const T* x_ptr, int32_t num_rows,
                                                      int32_t num_cols) {
  CUDA_1D_KERNEL_LOOP(j, num_cols) {
    T reduced = x_ptr[j];
    FOR_RANGE(int32_t, i, 1, num_rows) { reduced = binary_func(reduced, x_ptr[i * num_cols + j]); }
    y_ptr[j] = reduced;
  }
}

}  // namespace

struct RowOffsetFunctor final {
  OF_DEVICE_FUNC explicit RowOffsetFunctor(int32_t num_cols) : num_cols_(num_cols) {}
  OF_DEVICE_FUNC int32_t operator()(const int32_t& x) const { return x * num_cols_; }
  int32_t num_cols_;
};

template<typename T>
struct NdarrayScalarReduce<DeviceType::kGPU, T> final {
  static bool Matched(const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    return y.shape().ElemNum() == 1;
  }

  static void Reduce(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x,
                     const XpuVarNdarray<T>& tmp_storage) {
    CHECK(Matched(y, x));
    size_t x_size = x.shape().ElemNum();
    size_t tmp_storage_bytes = 0;
    auto DoReduce = [&](T* tmp_storage_ptr) {
      int retcode =
          hipcub::DeviceReduce::Reduce(tmp_storage_ptr, tmp_storage_bytes, x.ptr(), y.ptr(), x_size,
                                    hipcub::Sum(), ZeroVal<T>::value, ctx->hip_stream());
      CHECK_EQ(retcode, 0) << "hipcub::DeviceSegmentedReduce::Reduce error";
    };
    DoReduce(nullptr);
    CHECK_GE(tmp_storage.shape().ElemNum() * sizeof(T), tmp_storage_bytes);
    DoReduce(tmp_storage.ptr());
  }
};

template<typename T>
struct NdarrayMatrixRowReduce<DeviceType::kGPU, T> final {
  static bool Matched(const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    if (y.shape().ElemNum() > MaxVal<int32_t>()) { return false; }
    const auto& x_squeezed = SqueezeRight(x.shape());
    const auto& y_squeezed = SqueezeRight(y.shape());
    if (x_squeezed.NumAxes() == 0) { return false; }
    for (int i = 0; i < y_squeezed.NumAxes(); ++i) {
      if (x_squeezed.At(i) != y_squeezed.At(i)) { return false; }
    }
    CHECK_EQ(x.shape().ElemNum() % y.shape().ElemNum(), 0);
    return true;
  }

  static void Reduce(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x,
                     const XpuVarNdarray<T>& tmp_storage) {
    CHECK(Matched(y, x));
    int32_t num_rows = y.shape().ElemNum();
    int32_t num_cols = x.shape().ElemNum() / y.shape().ElemNum();
    RowOffsetFunctor get_row_offset(num_cols);
    hipcub::CountingInputIterator<int32_t> counting_intput_it(0);
    hipcub::TransformInputIterator<int32_t, RowOffsetFunctor, hipcub::CountingInputIterator<int32_t>>
        transform_input_iter(counting_intput_it, get_row_offset);
    size_t tmp_storage_bytes = 0;
    auto DoReduce = [&](T* tmp_storage_ptr) {
      int retcode = hipcub::DeviceSegmentedReduce::Reduce(
          tmp_storage_ptr, tmp_storage_bytes, x.ptr(), y.ptr(), num_rows, transform_input_iter,
          transform_input_iter + 1, hipcub::Sum(), ZeroVal<T>::value, ctx->hip_stream());
      CHECK_EQ(retcode, 0) << "hipcub::DeviceSegmentedReduce::Reduce error";
    };
    DoReduce(nullptr);
    CHECK_GE(tmp_storage.shape().ElemNum() * sizeof(T), tmp_storage_bytes);
    DoReduce(tmp_storage.ptr());
  }

 private:
  static XpuShape SqueezeRight(const XpuShape& shape) {
    std::vector<int64_t> dim_vec;
    for (int i = 0; i < shape.NumAxes(); ++i) { dim_vec.push_back(shape.At(i)); }
    for (int i = shape.NumAxes() - 1; i >= 0; --i) {
      if (dim_vec.at(i) != 1) { break; }
      dim_vec.pop_back();
    }
    if (dim_vec.empty()) { dim_vec.push_back(1LL); }
    return XpuShape(Shape(dim_vec));
  }
};

template<typename T>
struct NdarrayMatrixColReduce<DeviceType::kGPU, T> final {
  static bool Matched(const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    if (y.shape().ElemNum() > MaxVal<int32_t>()) { return false; }
    const auto& x_squeezed = SqueezeLeft(x.shape());
    const auto& y_squeezed = SqueezeLeft(y.shape());
    if (x_squeezed.NumAxes() == 0) { return false; }
    for (int i = 0; i < y_squeezed.NumAxes(); ++i) {
      if (x_squeezed.At(x_squeezed.NumAxes() - 1 - i)
          != y_squeezed.At(y_squeezed.NumAxes() - 1 - i)) {
        return false;
      }
    }
    CHECK_EQ(x.shape().ElemNum() % y.shape().ElemNum(), 0);
    return true;
  }

  static void Reduce(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x,
                     const XpuVarNdarray<T>& tmp_storage) {
    CHECK(Matched(y, x));
    int32_t num_rows = x.shape().ElemNum() / y.shape().ElemNum();
    int32_t num_cols = y.shape().ElemNum();
    RUN_CUDA_KERNEL((NdarrayMatrixColReduceNaiveCudaKernel<T, BinaryFuncAdd>), ctx, num_cols,
                    y.ptr(), x.ptr(), num_rows, num_cols);
  }

 private:
  static XpuShape SqueezeLeft(const XpuShape& shape) {
    std::vector<int64_t> dim_vec;
    bool all_squeezed = false;
    for (int i = 0; i < shape.NumAxes(); ++i) {
      if (all_squeezed == false) {
        if (shape.At(i) == 1) { continue; }
        all_squeezed = true;
      }
      dim_vec.push_back(shape.At(i));
    }
    if (dim_vec.empty()) { dim_vec.push_back(1LL); }
    return XpuShape(Shape(dim_vec));
  }
};

#define INSTANTIATE_NDARRAY_REDUCE_IMPL(type_cpp, type_proto)         \
  template struct NdarrayScalarReduce<DeviceType::kGPU, type_cpp>;    \
  template struct NdarrayMatrixRowReduce<DeviceType::kGPU, type_cpp>; \
  template struct NdarrayMatrixColReduce<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_NDARRAY_REDUCE_IMPL, ARITHMETIC_DATA_TYPE_SEQ);

}  // namespace oneflow
