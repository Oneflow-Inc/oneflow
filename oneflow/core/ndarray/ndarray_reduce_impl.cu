#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/ndarray/ndarray_reduce_impl.h"
#include "oneflow/core/ndarray/binary_func.h"
#include "oneflow/core/common/preprocessor.h"
#include "oneflow/core/common/shape.h"

namespace oneflow {

template<typename T, const T (*binary_func)(const T, const T), typename Enable = void>
struct CubFunctor4BianryFunc;

#define SPECIALIZE_CUB_FUNCTOR_4_BINARY_FUNC(binary_func, cub_functor)                            \
  template<typename T, const T (*bfunc)(const T, const T)>                                        \
  struct CubFunctor4BianryFunc<T, bfunc, typename std::enable_if<bfunc == &binary_func<T>>::type> \
      final {                                                                                     \
    using type = cub_functor;                                                                     \
  }

SPECIALIZE_CUB_FUNCTOR_4_BINARY_FUNC(BinaryFuncAdd, hipcub::Sum);
SPECIALIZE_CUB_FUNCTOR_4_BINARY_FUNC(BinaryFuncMax, hipcub::Max);
SPECIALIZE_CUB_FUNCTOR_4_BINARY_FUNC(BinaryFuncMin, hipcub::Min);

#undef SPECIALIZE_CUB_FUNCTOR_4_BINARY_FUNC

namespace {

template<typename T, const T (*binary_func)(const T, const T)>
void __global__ NdarrayMatrixColReduceNaiveCudaKernel(T* y_ptr, const T* x_ptr, int32_t num_rows,
                                                      int32_t num_cols) {
  CUDA_1D_KERNEL_LOOP(j, num_cols) {
    T reduced = x_ptr[j];
    FOR_RANGE(int32_t, i, 1, num_rows) { reduced = binary_func(reduced, x_ptr[i * num_cols + j]); }
    y_ptr[j] = reduced;
  }
}

}  // namespace

struct RowOffsetFunctor final {
  OF_DEVICE_FUNC explicit RowOffsetFunctor(int32_t num_cols) : num_cols_(num_cols) {}
  OF_DEVICE_FUNC int32_t operator()(const int32_t& x) const { return x * num_cols_; }
  int32_t num_cols_;
};

template<typename T, const T (*binary_func)(const T, const T)>
struct NdarrayScalarReduce<DeviceType::kGPU, T, binary_func> final {
  static bool Matched(const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    return y.shape().ElemNum() == 1;
  }

  static void Reduce(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x,
                     const XpuVarNdarray<T>& tmp_storage) {
    CHECK(Matched(y, x));
    size_t x_size = x.shape().ElemNum();
    size_t tmp_storage_bytes = 0;
    auto DoReduce = [&](T* tmp_storage_ptr) {
      int retcode =
          hipcub::DeviceReduce::Reduce(tmp_storage_ptr, tmp_storage_bytes, x.ptr(), y.ptr(), x_size,
                                    typename CubFunctor4BianryFunc<T, binary_func>::type(),
                                    UnitOfBinaryFunc<T, binary_func>::value, ctx->hip_stream());
      CHECK_EQ(retcode, 0) << "hipcub::DeviceSegmentedReduce::Reduce error";
    };
    DoReduce(nullptr);
    CHECK_GE(tmp_storage.shape().ElemNum() * sizeof(T), tmp_storage_bytes);
    DoReduce(tmp_storage.ptr());
  }
};

template<typename T, const T (*binary_func)(const T, const T)>
struct NdarrayMatrixRowReduce<DeviceType::kGPU, T, binary_func> final {
  static bool Matched(const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    if (y.shape().ElemNum() > GetMaxVal<int32_t>()) { return false; }
    const auto& x_squeezed = SqueezeRight(x.shape());
    const auto& y_squeezed = SqueezeRight(y.shape());
    if (x_squeezed.NumAxes() == 0) { return false; }
    for (int i = 0; i < y_squeezed.NumAxes(); ++i) {
      if (x_squeezed.At(i) != y_squeezed.At(i)) { return false; }
    }
    CHECK_EQ(x.shape().ElemNum() % y.shape().ElemNum(), 0);
    return true;
  }

  static void Reduce(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x,
                     const XpuVarNdarray<T>& tmp_storage) {
    CHECK(Matched(y, x));
    int32_t num_rows = y.shape().ElemNum();
    int32_t num_cols = x.shape().ElemNum() / y.shape().ElemNum();
    RowOffsetFunctor get_row_offset(num_cols);
    hipcub::CountingInputIterator<int32_t> counting_intput_it(0);
    hipcub::TransformInputIterator<int32_t, RowOffsetFunctor, hipcub::CountingInputIterator<int32_t>>
        transform_input_iter(counting_intput_it, get_row_offset);
    size_t tmp_storage_bytes = 0;
    auto DoReduce = [&](T* tmp_storage_ptr) {
      int retcode = hipcub::DeviceSegmentedReduce::Reduce(
          tmp_storage_ptr, tmp_storage_bytes, x.ptr(), y.ptr(), num_rows, transform_input_iter,
          transform_input_iter + 1, typename CubFunctor4BianryFunc<T, binary_func>::type(),
          UnitOfBinaryFunc<T, binary_func>::value, ctx->hip_stream());
      CHECK_EQ(retcode, 0) << "hipcub::DeviceSegmentedReduce::Reduce error";
    };
    DoReduce(nullptr);
    CHECK_GE(tmp_storage.shape().ElemNum() * sizeof(T), tmp_storage_bytes);
    DoReduce(tmp_storage.ptr());
  }

 private:
  static XpuShape SqueezeRight(const XpuShape& shape) {
    std::vector<int64_t> dim_vec;
    for (int i = 0; i < shape.NumAxes(); ++i) { dim_vec.push_back(shape.At(i)); }
    for (int i = shape.NumAxes() - 1; i >= 0; --i) {
      if (dim_vec.at(i) != 1) { break; }
      dim_vec.pop_back();
    }
    if (dim_vec.empty()) { dim_vec.push_back(1LL); }
    return XpuShape(Shape(dim_vec));
  }
};

template<typename T, const T (*binary_func)(const T, const T)>
struct NdarrayMatrixColReduce<DeviceType::kGPU, T, binary_func> final {
  static bool Matched(const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    if (y.shape().ElemNum() > GetMaxVal<int32_t>()) { return false; }
    const auto& x_squeezed = SqueezeLeft(x.shape());
    const auto& y_squeezed = SqueezeLeft(y.shape());
    if (x_squeezed.NumAxes() == 0) { return false; }
    for (int i = 0; i < y_squeezed.NumAxes(); ++i) {
      if (x_squeezed.At(x_squeezed.NumAxes() - 1 - i)
          != y_squeezed.At(y_squeezed.NumAxes() - 1 - i)) {
        return false;
      }
    }
    CHECK_EQ(x.shape().ElemNum() % y.shape().ElemNum(), 0);
    return true;
  }

  static void Reduce(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x,
                     const XpuVarNdarray<T>& tmp_storage) {
    CHECK(Matched(y, x));
    int32_t num_rows = x.shape().ElemNum() / y.shape().ElemNum();
    int32_t num_cols = y.shape().ElemNum();
    RUN_CUDA_KERNEL((NdarrayMatrixColReduceNaiveCudaKernel<T, binary_func>), ctx, num_cols, y.ptr(),
                    x.ptr(), num_rows, num_cols);
  }

 private:
  static XpuShape SqueezeLeft(const XpuShape& shape) {
    std::vector<int64_t> dim_vec;
    bool all_squeezed = false;
    for (int i = 0; i < shape.NumAxes(); ++i) {
      if (all_squeezed == false) {
        if (shape.At(i) == 1) { continue; }
        all_squeezed = true;
      }
      dim_vec.push_back(shape.At(i));
    }
    if (dim_vec.empty()) { dim_vec.push_back(1LL); }
    return XpuShape(Shape(dim_vec));
  }
};

#define INSTANTIATE_NDARRAY_REDUCE_IMPL(dtype, binary_func)                                       \
  template struct NdarrayScalarReduce<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype), binary_func>;    \
  template struct NdarrayMatrixRowReduce<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype), binary_func>; \
  template struct NdarrayMatrixColReduce<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype), binary_func>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_NDARRAY_REDUCE_IMPL, ARITHMETIC_DATA_TYPE_SEQ,
                                 REDUCE_BINARY_FUNC_SEQ);

}  // namespace oneflow
