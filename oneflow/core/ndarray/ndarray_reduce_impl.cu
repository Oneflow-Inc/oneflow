#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "oneflow/core/ndarray/ndarray_reduce_impl.h"
#include "oneflow/core/ndarray/binary_func.h"
#include "oneflow/core/common/preprocessor.h"
#include "oneflow/core/common/shape.h"
#include "oneflow/core/common/permutation_iterator.h"

namespace oneflow {

template<typename T, template<typename> class binary_func>
struct CubFunctor4BianryFunc;

#define SPECIALIZE_CUB_FUNCTOR_4_BINARY_FUNC(func_name)          \
  template<typename T>                                           \
  struct CubFunctor4BianryFunc<T, BinaryFunc##func_name> final { \
    using type = hipcub::func_name;                                 \
  };
OF_PP_FOR_EACH_ATOMIC(SPECIALIZE_CUB_FUNCTOR_4_BINARY_FUNC, REDUCE_BINARY_FUNC_NAME_SEQ);
#undef SPECIALIZE_CUB_FUNCTOR_4_BINARY_FUNC

namespace {

template<typename T, template<typename> class binary_func>
void __global__ NdarrayMatrixColReduceNaiveCudaKernel(T* y_ptr, const T* x_ptr, int32_t num_rows,
                                                      int32_t num_cols) {
  CUDA_1D_KERNEL_LOOP(j, num_cols) {
    T reduced = x_ptr[j];
    FOR_RANGE(int32_t, i, 1, num_rows) {
      reduced = binary_func<T>::Invoke(reduced, x_ptr[i * num_cols + j]);
    }
    y_ptr[j] = reduced;
  }
}

}  // namespace

struct RowOffsetFunctor final {
  OF_DEVICE_FUNC explicit RowOffsetFunctor(int32_t num_cols) : num_cols_(num_cols) {}
  OF_DEVICE_FUNC int32_t operator()(const int32_t& x) const { return x * num_cols_; }
  int32_t num_cols_;
};

template<typename T, template<typename> class binary_func>
struct NdarrayScalarReduce<DeviceType::kGPU, T, binary_func> final {
  static bool Matched(const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    return y.shape().ElemNum() == 1;
  }

  static void Reduce(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x,
                     const XpuVarNdarray<T>& tmp_storage) {
    CHECK(Matched(y, x));
    size_t x_size = x.shape().ElemNum();
    size_t tmp_storage_bytes = 0;
    auto DoReduce = [&](T* tmp_storage_ptr) {
      int retcode =
          hipcub::DeviceReduce::Reduce(tmp_storage_ptr, tmp_storage_bytes, x.ptr(), y.ptr(), x_size,
                                    typename CubFunctor4BianryFunc<T, binary_func>::type(),
                                    UnitOfBinaryFunc<T, binary_func>::Val(), ctx->hip_stream());
      CHECK_EQ(retcode, 0) << "hipcub::DeviceSegmentedReduce::Reduce error";
    };
    DoReduce(nullptr);
    CHECK_GE(tmp_storage.shape().ElemNum() * sizeof(T), tmp_storage_bytes);
    DoReduce(tmp_storage.ptr());
  }
};

template<typename T, template<typename> class binary_func>
struct NdarrayMatrixRowReduce<DeviceType::kGPU, T, binary_func> final {
  static bool Matched(const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    if (y.shape().ElemNum() > GetMaxVal<int32_t>()) { return false; }
    if (x.shape().NumAxes() != 2) { return false; }
    if (y.shape().NumAxes() != 2) { return false; }
    return x.shape().At(0) == y.shape().At(0) && y.shape().At(1) == 1;
  }

  static void Reduce(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x,
                     const XpuVarNdarray<T>& tmp_storage) {
    CHECK(Matched(y, x));
    int32_t num_rows = y.shape().ElemNum();
    int32_t num_cols = x.shape().ElemNum() / y.shape().ElemNum();
    RowOffsetFunctor get_row_offset(num_cols);
    hipcub::CountingInputIterator<int32_t> counting_intput_it(0);
    hipcub::TransformInputIterator<int32_t, RowOffsetFunctor, hipcub::CountingInputIterator<int32_t>>
        transform_input_iter(counting_intput_it, get_row_offset);
    size_t tmp_storage_bytes = 0;
    auto DoReduce = [&](T* tmp_storage_ptr) {
      int retcode = hipcub::DeviceSegmentedReduce::Reduce(
          tmp_storage_ptr, tmp_storage_bytes, x.ptr(), y.ptr(), num_rows, transform_input_iter,
          transform_input_iter + 1, typename CubFunctor4BianryFunc<T, binary_func>::type(),
          UnitOfBinaryFunc<T, binary_func>::Val(), ctx->hip_stream());
      CHECK_EQ(retcode, 0) << "hipcub::DeviceSegmentedReduce::Reduce error";
    };
    DoReduce(nullptr);
    CHECK_GE(tmp_storage.shape().ElemNum() * sizeof(T), tmp_storage_bytes);
    DoReduce(tmp_storage.ptr());
  }
};

template<typename T, template<typename> class binary_func>
struct NdarrayMatrixColReduce<DeviceType::kGPU, T, binary_func> final {
  static bool Matched(const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    if (y.shape().ElemNum() > GetMaxVal<int32_t>()) { return false; }
    if (x.shape().NumAxes() != 2) { return false; }
    if (y.shape().NumAxes() != 2) { return false; }
    return y.shape().At(0) == 1 && x.shape().At(1) == y.shape().At(1);
  }

  static void Reduce(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x,
                     const XpuVarNdarray<T>& tmp_storage) {
    CHECK(Matched(y, x));
    int32_t num_rows = x.shape().ElemNum() / y.shape().ElemNum();
    int32_t num_cols = y.shape().ElemNum();
    RUN_CUDA_KERNEL((NdarrayMatrixColReduceNaiveCudaKernel<T, binary_func>), ctx, num_cols, y.ptr(),
                    x.ptr(), num_rows, num_cols);
  }
};

template<typename T, template<typename> class binary_func>
struct NdarrayXYZCubeYReduce<DeviceType::kGPU, T, binary_func> final {
  static bool Matched(const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    return false;
    if (y.shape().ElemNum() > GetMaxVal<int32_t>()) { return false; }
    if (x.shape().NumAxes() != 3) { return false; }
    if (y.shape().NumAxes() != 3) { return false; }
    return x.shape().At(0) == y.shape().At(0) && y.shape().At(1) == 1
           && x.shape().At(2) == y.shape().At(2);
  }

  static void Reduce(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x,
                     const XpuVarNdarray<T>& tmp_storage) {
    CHECK(Matched(y, x));
    UNIMPLEMENTED();
  }
};

template<typename T, template<typename> class binary_func>
struct NdarrayXYZCubeXZReduce<DeviceType::kGPU, T, binary_func> final {
  static bool Matched(const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x) {
    if (y.shape().ElemNum() > GetMaxVal<int32_t>()) { return false; }
    if (x.shape().NumAxes() != 3) { return false; }
    if (y.shape().NumAxes() != 3) { return false; }
    return y.shape().At(0) == 1 && x.shape().At(1) == y.shape().At(1) && y.shape().At(2) == 1;
  }

  struct XYZ2YxzFunctor final {
    __host__ __device__ XYZ2YxzFunctor(int32_t dim_x, int32_t dim_y, int32_t dim_z)
        : dim_z_(dim_z), dim_xz_(dim_x * dim_z), dim_yz_(dim_y * dim_z) {}

    __host__ __device__ int32_t operator()(const int32_t& idx) const {
      const int32_t y = idx / dim_xz_;
      const int32_t xz_idx = idx % dim_xz_;
      const int32_t x = xz_idx / dim_z_;
      const int32_t z = xz_idx % dim_z_;
      return x * dim_yz_ + y * dim_z_ + z;
    }

    int32_t dim_z_;
    int32_t dim_xz_;
    int32_t dim_yz_;
  };

  static void Reduce(DeviceCtx* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const T>& x,
                     const XpuVarNdarray<T>& tmp_storage) {
    CHECK(Matched(y, x));
    int32_t num_rows = y.shape().ElemNum();
    int32_t num_cols = x.shape().ElemNum() / y.shape().ElemNum();

    RowOffsetFunctor get_row_offset(num_cols);
    hipcub::CountingInputIterator<int32_t> counting_intput_it(0);
    hipcub::TransformInputIterator<int32_t, RowOffsetFunctor, hipcub::CountingInputIterator<int32_t>>
        transform_input_iter(counting_intput_it, get_row_offset);

    XYZ2YxzFunctor xyz2yxz(x.shape().At(0), x.shape().At(1), x.shape().At(2));
    using XYZ2YxzIndexIter =
        hipcub::TransformInputIterator<int32_t, XYZ2YxzFunctor, hipcub::CountingInputIterator<int32_t>>;
    XYZ2YxzIndexIter xyz2yxz_iter(counting_intput_it, xyz2yxz);
    PermutationIterator<const T, const T*, XYZ2YxzIndexIter> x_iter(x.ptr(), xyz2yxz_iter);
    size_t tmp_storage_bytes = 0;
    auto DoReduce = [&](T* tmp_storage_ptr) {
      int retcode = hipcub::DeviceSegmentedReduce::Reduce(
          tmp_storage_ptr, tmp_storage_bytes, x_iter, y.ptr(), num_rows, transform_input_iter,
          transform_input_iter + 1, typename CubFunctor4BianryFunc<T, binary_func>::type(),
          UnitOfBinaryFunc<T, binary_func>::Val(), ctx->hip_stream());
      CHECK_EQ(retcode, 0) << "hipcub::DeviceSegmentedReduce::Reduce error";
    };
    DoReduce(nullptr);
    CHECK_GE(tmp_storage.shape().ElemNum() * sizeof(T), tmp_storage_bytes);
    DoReduce(tmp_storage.ptr());
  }
};

namespace {

template<typename T, int NDIMS, template<typename> class binary_func>
__global__ void NdarrayReduceGpuInplaceReduceAxis(const XpuReducedNdarray<T, NDIMS> dst_reduced,
                                                  const XpuReducedNdarray<T, NDIMS> x, int axis) {
  NdarrayReduceCore<T, NDIMS, binary_func>::ReduceAxis(dst_reduced, x, axis);
}

}  // namespace

template<typename T, int NDIMS, template<typename> class binary_func>
struct NdarrayReduceCoreWrapper<DeviceType::kGPU, T, NDIMS, binary_func> final {
  static void ReduceAxis(DeviceCtx* ctx, const XpuReducedNdarray<T, NDIMS>& dst_reduced,
                         const XpuReducedNdarray<T, NDIMS>& x, int axis) {
    size_t n = x.host_shape().HostElemNum();
    RUN_CUDA_KERNEL((NdarrayReduceGpuInplaceReduceAxis<T, NDIMS, binary_func>), ctx, n, dst_reduced,
                    x, axis);
  }
};

#define INSTANTIATE_NDARRAY_REDUCE_IMPL(dtype, binary_func)                                       \
  template struct NdarrayScalarReduce<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype), binary_func>;    \
  template struct NdarrayMatrixRowReduce<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype), binary_func>; \
  template struct NdarrayMatrixColReduce<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype), binary_func>; \
  template struct NdarrayXYZCubeYReduce<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype), binary_func>;  \
  template struct NdarrayXYZCubeXZReduce<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype), binary_func>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_NDARRAY_REDUCE_IMPL,
                                 ARITHMETIC_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ,
                                 REDUCE_BINARY_FUNC_SEQ);

#define INSTANTIATE_NDARRAY_REDUCE_CORE_WRAPPER(dtype_pair, NDIMS, binary_func)                   \
  template struct NdarrayReduceCoreWrapper<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype_pair), NDIMS, \
                                           binary_func>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_NDARRAY_REDUCE_CORE_WRAPPER,
                                 ARITHMETIC_DATA_TYPE_SEQ HALF_DATA_TYPE_SEQ, DIM_SEQ,
                                 REDUCE_BINARY_FUNC_SEQ);

}  // namespace oneflow
