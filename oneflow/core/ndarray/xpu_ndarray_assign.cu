#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ndarray/ndarray_assign_core.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"

namespace oneflow {

namespace {

template<typename T, typename X, int NDIMS>
__global__ void NdarrayAssignReducedGpu(XpuVarNdarray<T> y,
                                        const XpuReducedNdarray<X, NDIMS> reduced) {
  NdarrayAssignCore<T, X, NDIMS>::Assign(y, reduced);
}

template<typename T, typename X, int NDIMS>
__global__ void NdarrayAssignGpu(XpuVarNdarray<T> y, const XpuVarNdarray<const X> x) {
  NdarrayAssignCore<T, X, NDIMS>::Assign(y, x);
}

}  // namespace

template<typename T, typename X, int NDIMS>
struct NdarrayAssignCoreWrapper<DeviceType::kCUDA, T, X, NDIMS> final {
  static void Assign(ep::Stream* stream, XpuVarNdarray<T>* y,
                     const XpuReducedNdarray<X, NDIMS>& reduced) {
    size_t n = y->host_shape().HostElemNum();
    RUN_CUDA_KERNEL((NdarrayAssignReducedGpu<T, X, NDIMS>), stream, n, *y, reduced);
  }
  static void Assign(ep::Stream* ctx, const XpuVarNdarray<T>& y, const XpuVarNdarray<const X>& x) {
    size_t n = y.host_shape().HostElemNum();
    if (n == 0) { return; }
    RUN_CUDA_KERNEL((NdarrayAssignGpu<T, X, NDIMS>), ctx, n, y, x);
  }
};

#define INSTANTIATE_NDARRAY_ASSIGN(ret_dtype_pair, dtype_pair, NDIMS)                           \
  template struct NdarrayAssignCoreWrapper<DeviceType::kCUDA, OF_PP_PAIR_FIRST(ret_dtype_pair), \
                                           OF_PP_PAIR_FIRST(dtype_pair), NDIMS>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
    INSTANTIATE_NDARRAY_ASSIGN,
    ARITHMETIC_DATA_TYPE_SEQ UNSIGNED_INT_DATA_TYPE_SEQ BOOL_DATA_TYPE_SEQ,
    ARITHMETIC_DATA_TYPE_SEQ UNSIGNED_INT_DATA_TYPE_SEQ BOOL_DATA_TYPE_SEQ, DIM_SEQ);
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_NDARRAY_ASSIGN, HALF_DATA_TYPE_SEQ, HALF_DATA_TYPE_SEQ,
                                 DIM_SEQ);

}  // namespace oneflow
