#include "hip/hip_runtime.h"
#include "oneflow/core/ndarray/ndarray_reduce_core.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"

namespace oneflow {

namespace {

template<typename T, int NDIMS, const T (*binary_func)(const T, const T)>
__global__ void NdArrayReduceGpuImplaceReduceAxis(const XpuReducedNdarray<T, NDIMS> dst_reduced,
                                                  const XpuReducedNdarray<T, NDIMS> x, int axis) {
  NdArrayReduceCore<T, NDIMS, binary_func>::ReduceAxis(dst_reduced, x, axis);
}

}  // namespace

template<typename T, int NDIMS, const T (*binary_func)(const T, const T)>
struct NdArrayReduceCoreWrapper<DeviceType::kGPU, T, NDIMS, binary_func> final {
  static void ReduceAxis(DeviceCtx* ctx, const XpuReducedNdarray<T, NDIMS>& dst_reduced,
                         const XpuReducedNdarray<T, NDIMS>& x, int axis) {
    size_t n = x.host_shape().HostElemNum();
    RUN_CUDA_KERNEL((NdArrayReduceGpuImplaceReduceAxis<T, NDIMS, binary_func>), ctx, n, dst_reduced,
                    x, axis);
  }
};

#define INSTANTIATE_NDARRAY_REDUCE(dtype_pair, NDIMS, binary_func)                                \
  template struct NdArrayReduceCoreWrapper<DeviceType::kGPU, OF_PP_PAIR_FIRST(dtype_pair), NDIMS, \
                                           binary_func>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_NDARRAY_REDUCE, ARITHMETIC_DATA_TYPE_SEQ, DIM_SEQ,
                                 REDUCE_BINARY_FUNC_SEQ);

}  // namespace oneflow
