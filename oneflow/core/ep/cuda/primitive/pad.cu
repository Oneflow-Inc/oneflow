#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ep/include/primitive/pad.h"
#include "oneflow/core/ep/common/primitive/pad.h"
#include "oneflow/core/ep/cuda/primitive/type_seq.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include <hip/hip_runtime.h>

namespace oneflow {

namespace ep {

namespace primitive {

namespace {

template<typename T, int pack_size>
struct GetPackType {
  using type = typename std::aligned_storage<pack_size * sizeof(T), pack_size * sizeof(T)>::type;
};

template<typename T, int pack_size>
using PackType = typename GetPackType<T, pack_size>::type;

template<typename T, size_t pack_size>
union Pack {
  static_assert(sizeof(PackType<T, pack_size>) == sizeof(T) * pack_size, "");
  explicit __host__ __device__ Pack(T value) {
#pragma unroll
    for (int i = 0; i < pack_size; i++) { elem[i] = value; }
  }
  T elem[pack_size];
  PackType<T, pack_size> storage;
};

template<size_t num_dims, typename IndexType, typename T, int pack_size>
__global__ void PadKernel(PadParams<num_dims, IndexType> params, T pad_value) {
  IndexType global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  using LoadStoreType = PackType<T, pack_size>;
  const LoadStoreType* src = reinterpret_cast<const LoadStoreType*>(params.src);
  LoadStoreType* dst = reinterpret_cast<LoadStoreType*>(params.dst);
  IndexType src_index[num_dims];
  IndexType dst_index[num_dims];
  // for (IndexType linear_index = global_thread_id * pack_size; linear_index < params.elem_cnt;
  //      linear_index += gridDim.x * blockDim.x * pack_size) {
  for (IndexType linear_index = global_thread_id; linear_index < params.elem_cnt;
    linear_index += gridDim.x * blockDim.x) {
    params.dst_index_helper.OffsetToNdIndex(linear_index, dst_index);
    bool if_pad = false;
#pragma unroll
    for (int i = 0; i < num_dims; i++) {
      if (dst_index[i] >= params.padding_before[i]
          && dst_index[i] < params.out_size[i] - params.padding_after[i]) {
        src_index[i] = dst_index[i] - params.padding_before[i];
      } else {
        if_pad = true;
        break;
      }
    }
    if (!if_pad) {
      const IndexType src_offset = params.src_index_helper.NdIndexToOffset(src_index);
      dst[linear_index] = src[src_offset];
    } else {
      Pack<T, pack_size> packed_pad_val(pad_value);
      dst[linear_index] = packed_pad_val.storage;
    }
  }
}

template<typename T>
T GetValue(Scalar value) {
  return value.Value<T>();
}

template<>
half GetValue<half>(Scalar value) {
  return static_cast<half>(GetValue<float>(value));
}

#if CUDA_VERSION >= 11000

template<>
hip_bfloat16 GetValue<hip_bfloat16>(Scalar value) {
  return static_cast<hip_bfloat16>(GetValue<float>(value));
}

#endif  // CUDA_VERSION >= 11000

template<size_t max_pack_size>
size_t GetLaunchPackSize(size_t elem_size, size_t num_dims, void* dst, const int64_t* dst_dims,
                   const void* src, const int64_t* src_dims,
                   const int64_t* padding_before, const int64_t* padding_after) {
  static_assert(max_pack_size > 0 && (max_pack_size & (max_pack_size - 1)) == 0, "");
  CHECK_GT(elem_size, 0);
  CHECK_EQ((elem_size & (elem_size - 1)), 0);
  CHECK_EQ(max_pack_size % elem_size, 0);
  const int64_t last_dst_dim_size = dst_dims[num_dims - 1];
  const int64_t last_src_dim_size = src_dims[num_dims - 1];
  const int64_t last_padding_before_size = padding_before[num_dims - 1];
  const int64_t last_padding_after_size = padding_after[num_dims - 1];
  auto src_ptr = reinterpret_cast<std::uintptr_t>(src);
  auto dst_ptr = reinterpret_cast<std::uintptr_t>(dst);
  for (size_t size = max_pack_size; size > 1; size /= 2) {
    if (last_dst_dim_size % size == 0 && last_src_dim_size % size == 0
        && last_padding_before_size % size == 0 && last_padding_after_size % size == 0 
        && src_ptr % size == 0 && dst_ptr % size == 0) {
      return size;
    }
  }
  return 1;
}


template<size_t num_dims, typename IndexType, typename T, size_t pack_size>
void LaunchKernel(Stream* stream, PadParams<num_dims, IndexType> params, T pad_val) {
  hipStream_t hip_stream = stream->As<CudaStream>()->hip_stream();
  PadKernel<num_dims, IndexType, T, pack_size>
      <<<BlocksNum4ThreadsNum(params.elem_cnt), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params,
                                                                                           pad_val);
}

template<size_t num_dims, typename IndexType, typename T, size_t pack_size>
void LaunchKernel(Stream* stream, void* dst, const int64_t* dst_dims, const void* src,
                  const int64_t* src_dims, const int64_t* padding_before,
                  const int64_t* padding_after, T pad_val) {
  PadParams<num_dims, IndexType> params;
  params.dst_index_helper = NdIndexOffsetHelper<IndexType, num_dims>(dst_dims);
  params.src_index_helper = NdIndexOffsetHelper<IndexType, num_dims>(src_dims);
  params.dst = dst;
  params.src = src;
  size_t elem_cnt = 1;
  for (int i = 0; i < num_dims; i++) {
    params.padding_before[i] = padding_before[i];
    params.padding_after[i] = padding_after[i];
    params.out_size[i] = dst_dims[i];
    elem_cnt *= params.out_size[i];
  }
  // params.elem_cnt = elem_cnt * pack_size;
  params.elem_cnt = elem_cnt;
  printf("Elem count is: %ld \n", params.elem_cnt); 
  for(int i = 0; i < num_dims; i++){
    printf("padding before %d is: %ld \n", i, params.padding_before[i]); 
  }
  for(int i = 0; i < num_dims; i++){
    printf("padding after %d is: %ld \n", i, params.padding_after[i]); 
  }
  for(int i = 0; i < num_dims; i++){
    printf("outsize %d is: %ld \n", i, params.out_size[i]); 
  }
  LaunchKernel<num_dims, IndexType, T, pack_size>(stream, params, pad_val);
}

template<size_t num_dims, typename T, size_t pack_size>
void DispatchIndexType(Stream* stream, void* dst, const int64_t* dst_dims, const void* src,
                       const int64_t* src_dims, const int64_t* padding_before,
                       const int64_t* padding_after, T pad_val) {
  size_t elem_cnt = 1;
  for (size_t i = 0; i < num_dims; ++i) { elem_cnt *= dst_dims[i]; }
  if (elem_cnt < GetMaxVal<int32_t>()) {
    LaunchKernel<num_dims, int32_t, T, pack_size>(stream, dst, dst_dims, src, src_dims, padding_before,
                                       padding_after, pad_val);
  } else {
    LaunchKernel<num_dims, int64_t, T, pack_size>(stream, dst, dst_dims, src, src_dims, padding_before,
                                       padding_after, pad_val);
  }
}

constexpr int32_t Min(int32_t a, int32_t b) { return a < b ? a : b; }

template<typename T>
constexpr int32_t GetMaxPackSize(){
  return Min(128 / sizeof(T), 8); 
}

template<size_t num_dims, typename T>
void DispatchPackSize(Stream* stream, void* dst, const int64_t* dst_dims,
                      const void* src, const int64_t* src_dims, const int64_t* padding_before,
                      const int64_t* padding_after, T pad_val) {
  // max pack size wrong
  constexpr int32_t max_packsize = GetMaxPackSize<T>(); 
  size_t launch_pack_size = GetLaunchPackSize<max_packsize>(sizeof(T), num_dims, dst, dst_dims,
                                        src, src_dims,
                                        padding_before, padding_after); 
  int64_t simplify_dst_dims[num_dims]; 
  int64_t simplify_src_dims[num_dims]; 
  int64_t simplify_padding_before[num_dims]; 
  int64_t simplify_padding_after[num_dims]; 
  for(int i = 0; i < num_dims; i++){
    simplify_dst_dims[i] = dst_dims[i]; 
    simplify_src_dims[i] = src_dims[i]; 
    simplify_padding_before[i] = padding_before[i]; 
    simplify_padding_after[i] = padding_after[i]; 

    if(i == num_dims-1){
      simplify_dst_dims[i] /= launch_pack_size; 
      simplify_src_dims[i] /= launch_pack_size; 
      simplify_padding_before[i] /= launch_pack_size; 
      simplify_padding_after[i] /= launch_pack_size; 
    }
  }
  
  void (*func)(Stream* /*stream*/, void* /*dst*/, const int64_t* /*dst_dims*/, const void* /*src*/,
               const int64_t* /*src_dims*/, const int64_t* /*padding_before*/,
               const int64_t* /*padding_after*/, T) = nullptr;
  printf("Here launch pack size is: %ld \n", launch_pack_size); 
  if (launch_pack_size == 1) {
    func = DispatchIndexType<num_dims, T, 1>;
  } else if (launch_pack_size == 2) {
    func = DispatchIndexType<num_dims, T, 2>;
  } else if (launch_pack_size == 4) {
    func = DispatchIndexType<num_dims, T, 4>;
  } else if (launch_pack_size == 8) {
    func = DispatchIndexType<num_dims, T, 8>;
  } else if (launch_pack_size == 16) {
    func = DispatchIndexType<num_dims, T, 16>;
  } else {
    UNIMPLEMENTED();
  }
  // func(stream, dst, dst_dims, src, src_dims, padding_before, padding_after, pad_val);
  func(stream, dst, simplify_dst_dims, src, simplify_src_dims, simplify_padding_before, simplify_padding_after, pad_val);

}


template<typename T>
void LaunchWithSimplified(Stream* stream, size_t num_dims, void* dst, const int64_t* dst_dims,
                          const void* src, const int64_t* src_dims, const int64_t* padding_before,
                          const int64_t* padding_after, T pad_val) {
  void (*func)(Stream* /*stream*/, void* /*dst*/, const int64_t* /*dst_dims*/, const void* /*src*/,
               const int64_t* /*src_dims*/, const int64_t* /*padding_before*/,
               const int64_t* /*padding_after*/, T) = nullptr;
  if (num_dims == 1) {
    func = DispatchPackSize<1, T>;
  } else if (num_dims == 2) {
    func = DispatchPackSize<2, T>;
  } else if (num_dims == 3) {
    func = DispatchPackSize<3, T>;
  } else if (num_dims == 4) {
    func = DispatchPackSize<4, T>;
  } else if (num_dims == 5) {
    func = DispatchPackSize<5, T>;
  } else if (num_dims == 6) {
    func = DispatchPackSize<6, T>;
  } else if (num_dims == 7) {
    func = DispatchPackSize<7, T>;
  } else if (num_dims == 8) {
    func = DispatchPackSize<8, T>;
  } else {
    UNIMPLEMENTED();
  }
  func(stream, dst, dst_dims, src, src_dims, padding_before, padding_after, pad_val);
}

template<typename T>
class PadImpl : public Pad {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PadImpl);
  PadImpl() = default;
  ~PadImpl() override = default;

  void Launch(Stream* stream, size_t num_dims, void* dst, const int64_t* dst_dims, const void* src,
              const int64_t* src_dims, const int64_t* padding_before, const int64_t* padding_after,
              Scalar pad_val) override {
    LaunchWithSimplified<T>(stream, num_dims, dst, dst_dims, src, src_dims, padding_before,
                            padding_after, GetValue<T>(pad_val));
  }
};

template<typename T>
std::unique_ptr<Pad> NewPad() {
  return std::unique_ptr<Pad>(new PadImpl<T>());
}

#define CUDA_PAD_PRIMITIVE_TYPE_SEQ \
  CUDA_PRIMITIVE_INT32_TYPE_SEQ     \
  CUDA_PRIMITIVE_INT64_TYPE_SEQ     \
  CUDA_PRIMITIVE_FLOAT_TYPE_SEQ

class PadFactoryImpl : public PadFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PadFactoryImpl);
  PadFactoryImpl() = default;
  ~PadFactoryImpl() override = default;

  std::unique_ptr<Pad> New(DataType data_type) override {
#define MAKE_NEW_PAD_ENTRY(type_cpp, type_proto) {type_proto, NewPad<type_cpp>},

    static const std::map<DataType, std::function<std::unique_ptr<Pad>()>> new_pad_handle{
        OF_PP_FOR_EACH_TUPLE(MAKE_NEW_PAD_ENTRY, CUDA_PAD_PRIMITIVE_TYPE_SEQ)};

#undef MAKE_NEW_PAD_ENTRY

    const auto it = new_pad_handle.find(data_type);
    if (it != new_pad_handle.end()) {
      return it->second();
    } else {
      return nullptr;
    }
  }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kCUDA, PadFactory, PadFactoryImpl);

}  // namespace

}  // namespace primitive

}  // namespace ep

}  // namespace oneflow
