#include "hip/hip_runtime.h"
#include "oneflow/core/ep/include/primitive/pad.h"
#include "oneflow/core/ep/common/primitive/pad.h"
#include "oneflow/core/ep/cuda/primitive/type_seq.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include <hip/hip_runtime.h>

namespace oneflow { 

namespace ep {

namespace primitive {

namespace {

template<typename T, int pack_size>
struct GetPackType {
  using type = typename std::aligned_storage<pack_size * sizeof(T), pack_size * sizeof(T)>::type;
};

template<typename T, int pack_size>
using PackType = typename GetPackType<T, pack_size>::type;

template<typename T, size_t pack_size>
union Pack{
    static_assert(sizeof(PackType<T, pack_size>) == sizeof(T) * pack_size, "");
    explicit __host__ __device__ Pack(T value){
        #pragma unroll 
        for(int i = 0; i < pack_size; i++){
            elem[i] = value; 
        }
    }
    T elem[pack_size]; 
    PackType<T, pack_size> storage; 
}; 

template<size_t num_dims, typename IndexType, typename T, int pack_size>
__global__ void PadKernel(PadParams<num_dims, IndexType> params, T pad_value){
    IndexType global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    using LoadStoreType = PackType<T, pack_size>; 
    const LoadStoreType* src = reinterpret_cast<const LoadStoreType*>(params.src);
    LoadStoreType* dst = reinterpret_cast<LoadStoreType*>(params.dst);
    IndexType src_index[num_dims];
    IndexType dst_index[num_dims];
    for (IndexType linear_index = global_thread_id * pack_size; linear_index < params.elem_cnt;
        linear_index += gridDim.x * blockDim.x * pack_size){
        params.dst_index_helper.OffsetToNdIndex(linear_index, dst_index);
        bool if_pad = false; 
        #pragma unroll 
        for(int i = 0; i < num_dims; i++){
            if (dst_index[i] >= params.padding_before[i] && 
                dst_index[i] <= params.out_size[i] - params.padding_after[i]){
                    src_index[i] = dst_index[i] - params.padding_before[i]; 
                }
            else{ 
                if_pad = true; 
                break; 
            }
        }
        if(!if_pad){
            const IndexType src_offset = params.src_index_helper.NdIndexToOffset(src_index);
            dst[linear_index] = src[src_offset]; 
        } else {
            LoadStoreType packed_pad_val(pad_value);
            dst[linear_index] = packed_pad_val; 
        }
    }
}

template<typename T>
T GetValue(Scalar value) {
  return value.Value<T>();
}

template<>
half GetValue<half>(Scalar value) {
  return static_cast<half>(GetValue<float>(value));
}

#if CUDA_VERSION >= 11000

template<>
hip_bfloat16 GetValue<hip_bfloat16>(Scalar value) {
  return static_cast<hip_bfloat16>(GetValue<float>(value));
}

#endif  // CUDA_VERSION >= 11000

template<size_t num_dims, typename IndexType, typename T>
void LaunchKernel(Stream* stream, PadParams<num_dims, IndexType> params, T pad_val) {
  hipStream_t hip_stream = stream->As<CudaStream>()->hip_stream();
  PadKernel<num_dims, IndexType, T, /*pack_size*/1>
      <<<BlocksNum4ThreadsNum(params.elem_cnt), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params, pad_val);
}

template<typename T>
class PadImpl : public Pad {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PadImpl);
  PadImpl() = default;
  ~PadImpl() override = default;

  void Launch(Stream* stream, size_t num_dims, void* dst,
              const int64_t* dst_dims, const void* src,
              const int64_t* src_dims, const int64_t* padding_before,
              const int64_t* padding_after, Scalar pad_val) override {
        LaunchWithSimplified<T>(stream, num_dims, dst,
                                dst_dims, src,
                                src_dims, padding_before,
                                padding_after, GetValue<T>(pad_val)); 
    }
};

template<typename T>
std::unique_ptr<Pad> NewPad() {
  return std::unique_ptr<Pad>(new PadImpl<T>());
}

#define CUDA_PAD_PRIMITIVE_TYPE_SEQ \
  CUDA_PRIMITIVE_INT32_TYPE_SEQ     \
  CUDA_PRIMITIVE_INT64_TYPE_SEQ     \
  CUDA_PRIMITIVE_FLOAT_TYPE_SEQ     \
  CUDA_PRIMITIVE_DOUBLE_TYPE_SEQ    

class PadFactoryImpl : public PadFactory {
public:
 OF_DISALLOW_COPY_AND_MOVE(PadFactoryImpl);
 PadFactoryImpl() = default;
 ~PadFactoryImpl() override = default;

 std::unique_ptr<Pad> New(DataType data_type) override {
#define MAKE_NEW_PAD_ENTRY(type_cpp, type_proto) {type_proto, NewPad<type_cpp>},

   static const std::map<DataType, std::function<std::unique_ptr<Pad>()>> new_pad_handle{
       OF_PP_FOR_EACH_TUPLE(MAKE_NEW_PAD_ENTRY, CUDA_PAD_PRIMITIVE_TYPE_SEQ)};

#undef MAKE_NEW_PAD_ENTRY

   const auto it = new_pad_handle.find(data_type);
   if (it != new_pad_handle.end()) {
     return it->second();
   } else {
     return nullptr;
   }
 }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kCUDA, PadFactory, PadFactoryImpl);

} // namespace 

} // primitive 

} // namespace ep 

} // namespace oneflow 
