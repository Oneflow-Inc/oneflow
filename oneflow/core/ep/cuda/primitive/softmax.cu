/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ep/include/primitive/softmax.h"
#include "oneflow/core/ep/include/primitive/log_softmax.h"
#include "oneflow/core/ep/cuda/primitive/type_seq.h"
#include "oneflow/core/cuda/softmax.cuh"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace ep {
namespace primitive {

namespace {

enum class Algorithm {
  kSoftmax,
  kLogSoftmax,
};

template<Algorithm algorithm, typename T>
void SoftmaxGpu(hipStream_t hip_stream, size_t rows, size_t cols, const T* x, T* y) {
  using ComputeType = typename cuda::softmax::DefaultComputeType<T>::type;
  oneflow::cuda::softmax::DirectLoad<T, ComputeType> load(x, cols);
  oneflow::cuda::softmax::DirectStore<ComputeType, T> store(y, cols);
  if (algorithm == Algorithm::kSoftmax) {
    OF_CUDA_CHECK((cuda::softmax::DispatchSoftmax<decltype(load), decltype(store), ComputeType>(
        hip_stream, load, store, rows, cols)));
  } else if (algorithm == Algorithm::kLogSoftmax) {
    OF_CUDA_CHECK((cuda::softmax::DispatchLogSoftmax<decltype(load), decltype(store), ComputeType>(
        hip_stream, load, store, rows, cols)));
  } else {
    UNIMPLEMENTED();
  }
}

template<typename SoftmaxBase, Algorithm algorithm, typename T>
class SoftmaxImpl : public SoftmaxBase {
 public:
  OF_DISALLOW_COPY_AND_MOVE(SoftmaxImpl);
  SoftmaxImpl() = default;
  ~SoftmaxImpl() override = default;

  void Launch(Stream* stream, size_t rows, size_t cols, const void* x, void* y) override {
    hipStream_t hip_stream = stream->As<CudaStream>()->hip_stream();
    SoftmaxGpu<algorithm, T>(hip_stream, rows, cols, reinterpret_cast<const T*>(x),
                             reinterpret_cast<T*>(y));
  }
};

template<typename SoftmaxBase, Algorithm algorithm, typename T>
std::unique_ptr<SoftmaxBase> NewSoftmax() {
  return std::unique_ptr<SoftmaxBase>(new SoftmaxImpl<SoftmaxBase, algorithm, T>());
}

template<typename FactoryBase, typename SoftmaxBase, Algorithm algorithm>
class GenericSoftmaxFactoryImpl : public FactoryBase {
 public:
  OF_DISALLOW_COPY_AND_MOVE(GenericSoftmaxFactoryImpl);
  GenericSoftmaxFactoryImpl() = default;
  ~GenericSoftmaxFactoryImpl() override = default;

  std::unique_ptr<SoftmaxBase> New(DataType data_type) override {
#define MAKE_NEW_SOFTMAX_ENTRY(type_cpp, type_proto) \
  {type_proto, NewSoftmax<SoftmaxBase, algorithm, type_cpp>},

    static const std::map<DataType, std::function<std::unique_ptr<SoftmaxBase>()>>
        new_softmax_handle{
            OF_PP_FOR_EACH_TUPLE(MAKE_NEW_SOFTMAX_ENTRY, CUDA_PRIMITIVE_FLOATING_TYPE_SEQ)};

#undef MAKE_NEW_SOFTMAX_ENTRY

    const auto it = new_softmax_handle.find(data_type);
    if (it != new_softmax_handle.end()) {
      return it->second();
    } else {
      return nullptr;
    }
  }
};

using SoftmaxFactoryImpl = GenericSoftmaxFactoryImpl<SoftmaxFactory, Softmax, Algorithm::kSoftmax>;
using LogSoftmaxFactoryImpl =
    GenericSoftmaxFactoryImpl<LogSoftmaxFactory, LogSoftmax, Algorithm::kLogSoftmax>;
REGISTER_PRIMITIVE_FACTORY(DeviceType::kCUDA, SoftmaxFactory, SoftmaxFactoryImpl);
REGISTER_PRIMITIVE_FACTORY(DeviceType::kCUDA, LogSoftmaxFactory, LogSoftmaxFactoryImpl);

}  // namespace

}  // namespace primitive
}  // namespace ep

}  // namespace oneflow
