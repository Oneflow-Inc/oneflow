#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ep/include/primitive/permute.h"
#include "oneflow/core/ep/common/primitive/permute_impl.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include <hip/hip_runtime.h>

namespace oneflow {

namespace ep {
namespace primitive {

namespace permute {

namespace internal {

namespace {

constexpr int32_t kMov4TileSize = 32;
constexpr int32_t kMov2TileSize = 64;
constexpr int32_t kBlockRows = 8;

template<size_t num_dims, size_t movement_size, typename IndexType>
__global__ void PermuteKernel(PermuteKernelParams<num_dims, IndexType> params) {
  using T = typename std::aligned_storage<movement_size, movement_size>::type;
  const T* src = reinterpret_cast<const T*>(params.src);
  T* dst = reinterpret_cast<T*>(params.dst);
  IndexType src_index[num_dims];
  IndexType dst_index[num_dims];
  CUDA_1D_KERNEL_LOOP_T(IndexType, i, params.count) {
    params.dst_index_helper.OffsetToNdIndex(i, dst_index);
#pragma unroll
    for (size_t dim = 0; dim < num_dims; ++dim) {
      src_index[params.permutation[dim]] = dst_index[dim];
    }
    IndexType src_offset = params.src_index_helper.NdIndexToOffset(src_index);
    dst[i] = src[src_offset];
  }
}

// (B, X, Y) -> (B, Y, X)
// refer from https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
template<size_t num_dims, size_t movement_size, size_t tile_size, typename IndexType>
__global__ void BatchTransposeKernel(const void* src_ptr, void* dst_ptr, IndexType rows,
                                     IndexType cols, IndexType num_tile_rows,
                                     IndexType num_tile_cols, int32_t block_nums) {
  const IndexType src_rows = rows;
  const IndexType src_cols = cols;
  const IndexType dst_rows = cols;
  const IndexType dst_cols = rows;

  using T = typename std::aligned_storage<movement_size, movement_size>::type;
  __shared__ T tile[tile_size][tile_size + 1];  // To avoid bank conflict.

  const T* src = reinterpret_cast<const T*>(src_ptr);
  T* dst = reinterpret_cast<T*>(dst_ptr);

  IndexType batch_num_tile = num_tile_rows * num_tile_cols;
  for (int i = blockIdx.x, step = gridDim.x; i < block_nums; i += step) {
    const IndexType batch_index = i / batch_num_tile;  // the index of batch.
    const IndexType tile_index =
        i - batch_index * batch_num_tile;  // equal to i % (num_tile_rows*num_tile_cols). the
                                           // flatten index of tile in a batch.

    const IndexType tile_row_index =
        tile_index / num_tile_cols;  // the row index of tile in a batch.
    const IndexType tile_col_index =
        tile_index
        - tile_row_index
              * num_tile_cols;  // equal to k % num_tile_cols. the col index of tile in a batch.

    const IndexType offset = batch_index * src_rows * src_cols;
    {
      IndexType col_in_tile = threadIdx.x;
      IndexType col_in_matrix = tile_col_index * tile_size + threadIdx.x;
#pragma unroll
      for (IndexType row_in_tile = threadIdx.y; row_in_tile < tile_size;
           row_in_tile += kBlockRows) {
        IndexType row_in_matrix = row_in_tile + tile_row_index * tile_size;
        if (col_in_matrix < src_cols && row_in_matrix < src_rows) {
          tile[row_in_tile][col_in_tile] = src[offset + row_in_matrix * src_cols + col_in_matrix];
        }
      }
    }
    __syncthreads();
    {
      IndexType col_in_tile = threadIdx.x;
      IndexType col_in_matrix = tile_row_index * tile_size + threadIdx.x;
#pragma unroll
      for (IndexType row_in_tile = threadIdx.y; row_in_tile < tile_size;
           row_in_tile += kBlockRows) {
        IndexType row_in_matrix = row_in_tile + tile_col_index * tile_size;
        if (col_in_matrix < dst_cols && row_in_matrix < dst_rows) {
          dst[offset + row_in_matrix * dst_cols + col_in_matrix] = tile[col_in_tile][row_in_tile];
        }
      }
    }
    __syncthreads();
  }
}

/*
Here is a Movementsie=2 version of Batch Transpose.
When the H W can be divided by 2. we can read data use movementsize=4, and write back as
movementsize=4.
*/
template<size_t num_dims, size_t tile_size, typename IndexType>
__global__ void BatchTransposeMovement2Kernel(const void* src_ptr, void* dst_ptr, IndexType rows,
                                              IndexType cols, IndexType num_tile_rows,
                                              IndexType num_tile_cols, int32_t block_nums) {
  const IndexType src_rows = rows;
  const IndexType src_cols = cols;
  const IndexType dst_rows = cols;
  const IndexType dst_cols = rows;

  static_assert(tile_size % 2 == 0, "");
  using T_MOV2 = typename std::aligned_storage<2, 2>::type;
  using T_MOV4 = typename std::aligned_storage<4, 4>::type;

  const T_MOV4* src = reinterpret_cast<const T_MOV4*>(src_ptr);
  T_MOV4* dst = reinterpret_cast<T_MOV4*>(dst_ptr);

  // Use union structure to process Load and Store.
  __shared__ union {
    T_MOV2 tile_m2[tile_size][tile_size + 2];      // half [64][66]
    T_MOV4 tile_m4[tile_size][tile_size / 2 + 1];  // half2 [64][33]
  } tile_mem;

  IndexType batch_num_tile = num_tile_rows * num_tile_cols;
  for (int i = blockIdx.x, step = gridDim.x; i < block_nums; i += step) {
    const IndexType batch_index = i / batch_num_tile;  // the index of batch.
    const IndexType tile_index =
        i - batch_index * batch_num_tile;  // equal to i % (num_tile_rows*num_tile_cols). the
                                           // flatten index of tile in a batch.

    const IndexType tile_row_index =
        tile_index / num_tile_cols;  // the row index of tile in a batch.
    const IndexType tile_col_index =
        tile_index
        - tile_row_index
              * num_tile_cols;  // equal to k % num_tile_cols. the col index of tile in a batch.

    const IndexType offset = batch_index * src_rows * src_cols;
    {
      IndexType col_in_tile = threadIdx.x;
      IndexType col_in_matrix = tile_col_index * tile_size + threadIdx.x * 2;
#pragma unroll
      for (IndexType row_in_tile = threadIdx.y; row_in_tile < tile_size;
           row_in_tile += kBlockRows) {
        IndexType row_in_matrix = row_in_tile + tile_row_index * tile_size;
        if (col_in_matrix < src_cols && row_in_matrix < src_rows) {
          tile_mem.tile_m4[row_in_tile][col_in_tile] =
              src[(offset + row_in_matrix * src_cols + col_in_matrix) / 2];
        }
      }
    }
    __syncthreads();
    {
      IndexType col_in_tile = threadIdx.x;
      IndexType col_in_matrix = tile_row_index * tile_size + threadIdx.x * 2;
#pragma unroll
      for (IndexType row_in_tile = threadIdx.y; row_in_tile < tile_size;
           row_in_tile += kBlockRows) {
        IndexType row_in_matrix = row_in_tile + tile_col_index * tile_size;
        union {
          T_MOV4 m4;
          T_MOV2 m2[2];
        } tmp_storage;

        if (col_in_matrix < dst_cols && row_in_matrix < dst_rows) {
          tmp_storage.m2[0] = tile_mem.tile_m2[col_in_tile * 2][row_in_tile];
          tmp_storage.m2[1] = tile_mem.tile_m2[col_in_tile * 2 + 1][row_in_tile];
          dst[(offset + row_in_matrix * dst_cols + col_in_matrix) / 2] = tmp_storage.m4;
        }
      }
    }
    __syncthreads();
  }
}

template<size_t num_dims, size_t movement_size, size_t tile_size, typename IndexType>
void LaunchBatchTransposeKernel(hipStream_t& hip_stream,
                                const PermuteKernelParams<num_dims, IndexType>& params,
                                const IndexType& num_batches, const IndexType& rows,
                                const IndexType& cols) {
  IndexType num_tile_rows = (rows + tile_size - 1) / tile_size;
  IndexType num_tile_cols = (cols + tile_size - 1) / tile_size;
  const int32_t block_nums = num_batches * num_tile_rows * num_tile_cols;
  int32_t launched_block_nums = std::min(block_nums, kCudaMaxBlocksNum);
  if (tile_size == kMov2TileSize) {
    const int32_t half2_thread = tile_size / 2;  // cause each thread process two half elements.
    BatchTransposeMovement2Kernel<num_dims, kMov2TileSize, IndexType>
        <<<launched_block_nums, dim3(half2_thread, kBlockRows), 0, hip_stream>>>(
            params.src, params.dst, rows, cols, num_tile_rows, num_tile_cols,
            block_nums);  // Set threads num as 32x8 cause each threads
                          // process 4 elements to 64x66 half share memory.
  } else {
    BatchTransposeKernel<num_dims, movement_size, tile_size, IndexType>
        <<<launched_block_nums, dim3(tile_size, kBlockRows), 0, hip_stream>>>(
            params.src, params.dst, rows, cols, num_tile_rows, num_tile_cols, block_nums);
  }
}

template<size_t tile_size, typename IndexType>
bool CheckIfGreaterEqualThanTileSize(const IndexType& rows, const IndexType& cols) {
  if (rows < tile_size || cols < tile_size) { return false; }
  return true;
}

template<size_t num_dims, size_t tile_size, typename IndexType>
bool CheckLaunchBatchTranspose(const int* permutation, const IndexType& num_batches,
                               const IndexType& rows, const IndexType& cols) {
  if (CheckIfGreaterEqualThanTileSize<tile_size, IndexType>(rows, cols)) {
    if (num_batches == 1 && permutation[1] == 0 && permutation[0] == 1) {
      // 2d tensor case: (0, 1) -> (1, 0)
      return true;
    } else if (num_dims == 3 && permutation[2] == 1 && permutation[1] == 2) {
      // 3d tensor case: (0, 1, 2) -> (0, 2, 1)
      return true;
    } else {
      return false;
    }
  }
  return false;
}

template<typename IndexType, size_t movement_size>
bool CheckUseMov2(const IndexType& rows, const IndexType& cols, const void* src, void* dst) {
  auto src_ptr = reinterpret_cast<std::uintptr_t>(src);
  auto dst_ptr = reinterpret_cast<std::uintptr_t>(dst);
  return (movement_size == 2) && (rows % 2 == 0) && (cols % 2 == 0) && (src_ptr % 4 == 0)
         && (dst_ptr % 4 == 0);
}

template<size_t num_dims, typename IndexType>
void InferBatchTransposeShape(const int64_t* src_dims, IndexType* num_batches, IndexType* rows,
                              IndexType* cols) {
  if (num_dims == 2) {
    *num_batches = 1;
    *rows = src_dims[0];
    *cols = src_dims[1];
  } else {
    *num_batches = src_dims[0];
    *rows = src_dims[1];
    *cols = src_dims[2];
  }
}

template<size_t num_dims, size_t movement_size, typename IndexType>
void LaunchKernel(Stream* stream, const int64_t* src_dims, const void* src, const int* permutation,
                  void* dst, size_t count) {
  PermuteKernelParams<num_dims, IndexType> params =
      MakePermuteParams<num_dims, IndexType>(src_dims, src, permutation, dst, count);
  hipStream_t hip_stream = stream->As<CudaStream>()->hip_stream();

  if (num_dims == 2 || num_dims == 3) {
    IndexType num_batches;
    IndexType rows;
    IndexType cols;
    InferBatchTransposeShape<num_dims, IndexType>(src_dims, &num_batches, &rows, &cols);
    if (CheckLaunchBatchTranspose<num_dims, kMov4TileSize>(params.permutation, num_batches, rows,
                                                           cols)) {
      if (CheckUseMov2<IndexType, movement_size>(rows, cols, src, dst)) {
        LaunchBatchTransposeKernel<num_dims, 2, kMov2TileSize, IndexType>(hip_stream, params,
                                                                          num_batches, rows, cols);
      } else {
        LaunchBatchTransposeKernel<num_dims, movement_size, kMov4TileSize, IndexType>(
            hip_stream, params, num_batches, rows, cols);
      }
    } else {
      if (params.count == 0) { return; }
      PermuteKernel<num_dims, movement_size, IndexType>
          <<<BlocksNum4ThreadsNum(params.count), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params);
    }
  } else {
    if (params.count == 0) { return; }
    PermuteKernel<num_dims, movement_size, IndexType>
        <<<BlocksNum4ThreadsNum(params.count), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params);
  }
}

class PermuteImpl : public Permute {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteImpl);
  PermuteImpl() = default;
  ~PermuteImpl() override = default;

  using Permute::Launch;
  void Launch(Stream* stream, DataType data_type, size_t num_dims, const int64_t* src_dims,
              const void* src, const int* permutation, void* dst) override {
    SimplifyThenLaunch(stream, data_type, num_dims, src_dims, src, permutation, dst);
  }
};

class PermuteFactoryImpl : public PermuteFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(PermuteFactoryImpl);
  PermuteFactoryImpl() = default;
  ~PermuteFactoryImpl() override = default;

  std::unique_ptr<Permute> New(size_t max_num_dims) override {
    if (max_num_dims <= kMaxNumDims) {
      return std::unique_ptr<Permute>(new PermuteImpl());
    } else {
      return nullptr;
    }
  }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kCUDA, PermuteFactory, PermuteFactoryImpl);

}  // namespace

}  // namespace internal

}  // namespace permute

}  // namespace primitive
}  // namespace ep

}  // namespace oneflow
