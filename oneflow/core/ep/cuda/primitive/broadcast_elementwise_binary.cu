/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ep/include/primitive/broadcast_elementwise_binary.h"
#include "oneflow/core/ep/common/primitive/broadcast_elementwise_binary.h"
#include "oneflow/core/ep/cuda/primitive/type_seq.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/ep/cuda/primitive/binary_functor.cuh"

namespace oneflow {

namespace ep {
namespace primitive {
namespace broadcast_elementwise_binary {

template<BinaryOp binary_op, typename Src, typename Dst>
std::unique_ptr<BroadcastElementwiseBinary> NewBroadcastElementwiseBinary(Scalar attr0,
                                                                          Scalar attr1);

namespace {

class BroadcastElementwiseBinaryFactoryImpl : public BroadcastElementwiseBinaryFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(BroadcastElementwiseBinaryFactoryImpl);
  BroadcastElementwiseBinaryFactoryImpl() = default;
  ~BroadcastElementwiseBinaryFactoryImpl() override = default;

  std::unique_ptr<BroadcastElementwiseBinary> New(BinaryOp op, DataType src_type, DataType dst_type,
                                                  size_t max_num_dims) override {
    return New(op, src_type, dst_type, max_num_dims, Scalar(), Scalar());
  }

  std::unique_ptr<BroadcastElementwiseBinary> New(BinaryOp op, DataType src_type, DataType dst_type,
                                                  size_t max_num_dims, Scalar attr0) override {
    return New(op, src_type, dst_type, max_num_dims, attr0, Scalar());
  }

  std::unique_ptr<BroadcastElementwiseBinary> New(BinaryOp binary_op, DataType src_type,
                                                  DataType dst_type, size_t max_num_dims,
                                                  Scalar attr0, Scalar attr1) override {
    if (max_num_dims > kMaxNumDims) { return nullptr; }
#define MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_MATH_ENTRY(binary_op, data_type_pair) \
  {std::make_tuple(binary_op, OF_PP_PAIR_SECOND(data_type_pair),                    \
                   OF_PP_PAIR_SECOND(data_type_pair)),                              \
   NewBroadcastElementwiseBinary<binary_op, OF_PP_PAIR_FIRST(data_type_pair),       \
                                 OF_PP_PAIR_FIRST(data_type_pair)>},

#define MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_COMPARASION_AND_LOGICAL_ENTRY(      \
    binary_op, src_data_type_pair, dst_data_type_pair)                            \
  {std::make_tuple(binary_op, OF_PP_PAIR_SECOND(src_data_type_pair),              \
                   OF_PP_PAIR_SECOND(dst_data_type_pair)),                        \
   NewBroadcastElementwiseBinary<binary_op, OF_PP_PAIR_FIRST(src_data_type_pair), \
                                 OF_PP_PAIR_FIRST(dst_data_type_pair)>},

#define MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_ACTIVATION_GRAD_ENTRY(binary_op, data_type_pair) \
  {std::make_tuple(binary_op, OF_PP_PAIR_SECOND(data_type_pair),                               \
                   OF_PP_PAIR_SECOND(data_type_pair)),                                         \
   NewBroadcastElementwiseBinary<binary_op, OF_PP_PAIR_FIRST(data_type_pair),                  \
                                 OF_PP_PAIR_FIRST(data_type_pair)>},

    static const std::map<
        std::tuple<BinaryOp, DataType, DataType>,
        std::function<std::unique_ptr<BroadcastElementwiseBinary>(Scalar, Scalar)>>
        new_broadcast_elementwise_binary_handle{
            OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_MATH_ENTRY,
                                             BINARY_MATH_OP_SEQ, CUDA_PRIMITIVE_ALL_TYPE_SEQ)

                OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
                    MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_COMPARASION_AND_LOGICAL_ENTRY,
                    BINARY_COMPARISION_OP_SEQ BINARY_LOGICAL_OP_SEQ, CUDA_PRIMITIVE_ALL_TYPE_SEQ,
                    CUDA_PRIMITIVE_BOOL_TYPE_SEQ)

                    OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
                        MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_ACTIVATION_GRAD_ENTRY,
                        BINARY_ACTIVATION_BACKWARD_OP_SEQ, CUDA_PRIMITIVE_FLOATING_TYPE_SEQ)

                        OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
                            MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_ACTIVATION_GRAD_ENTRY,
                            BINARY_MATH_BACKWARD_OP_SEQ, CUDA_PRIMITIVE_FLOATING_TYPE_SEQ)

                            OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
                                MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_MATH_ENTRY,
                                BINARY_BITWISE_OP_SEQ,
                                CUDA_PRIMITIVE_INT_TYPE_SEQ CUDA_PRIMITIVE_BOOL_TYPE_SEQ)};

#undef MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_COMPARASION_AND_LOGICAL_ENTRY
#undef MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_MATH_ENTRY

    const auto it = new_broadcast_elementwise_binary_handle.find(
        std::make_tuple(binary_op, src_type, dst_type));
    if (it != new_broadcast_elementwise_binary_handle.end()) {
      return it->second(attr0, attr1);
    } else {
      return nullptr;
    }
  }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kCUDA, BroadcastElementwiseBinaryFactory,
                           BroadcastElementwiseBinaryFactoryImpl);
}  // namespace
}  // namespace broadcast_elementwise_binary
}  // namespace primitive
}  // namespace ep

}  // namespace oneflow
