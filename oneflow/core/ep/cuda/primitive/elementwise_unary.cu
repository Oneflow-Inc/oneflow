/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ep/common/primitive/elementwise_unary.h"
#include "oneflow/core/ep/cuda/primitive/unary_functor.cuh"

namespace oneflow {

namespace ep {
namespace primitive {

namespace {

template<UnaryOp unary_op, typename Src, typename Dst>
class ElementwiseUnaryImpl : public ElementwiseUnary {
 public:
  OF_DISALLOW_COPY_AND_MOVE(ElementwiseUnaryImpl);
  ElementwiseUnaryImpl(Scalar attr0, Scalar attr1) : attr0(attr0), attr1(attr1) {}
  ~ElementwiseUnaryImpl() override = default;

  void Launch(Stream* stream, const void* src, void* dst, size_t count) override {
    auto* hip_stream = stream->As<CudaStream>();
    auto functor = UnaryFunctor<DeviceType::kCUDA, unary_op, Dst, Src>(attr0, attr1);
    OF_CUDA_CHECK((cuda::elementwise::Unary<decltype(functor), Dst, Src>(
        functor, count, reinterpret_cast<Dst*>(dst), reinterpret_cast<const Src*>(src),
        hip_stream->hip_stream())));
  }

 protected:
  Scalar attr0, attr1;
};

template<UnaryOp unary_op, typename Src, typename Dst>
std::unique_ptr<ElementwiseUnary> NewElementwiseUnary(Scalar attr0, Scalar attr1) {
  return std::unique_ptr<ElementwiseUnary>(
      new ElementwiseUnaryImpl<unary_op, Src, Dst>(attr0, attr1));
}

class ElementwiseUnaryFactoryImpl : public ElementwiseUnaryFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(ElementwiseUnaryFactoryImpl);
  ElementwiseUnaryFactoryImpl() = default;
  ~ElementwiseUnaryFactoryImpl() override = default;

  std::unique_ptr<ElementwiseUnary> New(UnaryOp unary_op, DataType src_type,
                                        DataType dst_dtype) override {
    return New(unary_op, src_type, dst_dtype, Scalar(), Scalar());
  }

  std::unique_ptr<ElementwiseUnary> New(UnaryOp unary_op, DataType src_type, DataType dst_dtype,
                                        Scalar attr0) override {
    return New(unary_op, src_type, dst_dtype, attr0, Scalar());
  }

  std::unique_ptr<ElementwiseUnary> New(UnaryOp unary_op, DataType src_type, DataType dst_dtype,
                                        Scalar attr0, Scalar attr1) override {
#define MAKE_NEW_SAME_DTYPE_ELEMENTWISE_UNARY_ENTRY(unary_op, dtype_pair)                   \
  {std::make_tuple(unary_op, OF_PP_PAIR_SECOND(dtype_pair), OF_PP_PAIR_SECOND(dtype_pair)), \
   NewElementwiseUnary<unary_op, OF_PP_PAIR_FIRST(dtype_pair), OF_PP_PAIR_FIRST(dtype_pair)>},

#define MAKE_NEW_DIFFERENT_DTYPE_ELEMENTWISE_UNARY_ENTRY(unary_op, src_type_pair, dst_dtype_pair)  \
  {std::make_tuple(unary_op, OF_PP_PAIR_SECOND(src_type_pair), OF_PP_PAIR_SECOND(dst_dtype_pair)), \
   NewElementwiseUnary<unary_op, OF_PP_PAIR_FIRST(src_type_pair),                                  \
                       OF_PP_PAIR_FIRST(dst_dtype_pair)>},

    static const std::map<std::tuple<UnaryOp, DataType, DataType>,
                          std::function<std::unique_ptr<ElementwiseUnary>(Scalar, Scalar)>>
        new_elementwise_unary_handle{
            // For All Type OP
            OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_NEW_SAME_DTYPE_ELEMENTWISE_UNARY_ENTRY,
                                             UNARY_MATH_OP_SEQ, CUDA_PRIMITIVE_ALL_TYPE_SEQ)
            // For Float Type OP
            OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_NEW_SAME_DTYPE_ELEMENTWISE_UNARY_ENTRY,
                                             UNARY_FLOATING_MATH_OP_SEQ,
                                             CUDA_PRIMITIVE_FLOATING_TYPE_SEQ)

            // For Int Type OP
            OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_NEW_SAME_DTYPE_ELEMENTWISE_UNARY_ENTRY,
                                             UNARY_INT_MATH_OP_SEQ, CUDA_PRIMITIVE_INT_TYPE_SEQ)

            // For Utils OP
            OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_NEW_DIFFERENT_DTYPE_ELEMENTWISE_UNARY_ENTRY,
                                             UNARY_UTILS_OP_SEQ, UTIL_OPS_DATA_TYPE_SEQ,
                                             CUDA_PRIMITIVE_BOOL_TYPE_SEQ)

            // For Logical OP
            OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_NEW_DIFFERENT_DTYPE_ELEMENTWISE_UNARY_ENTRY,
                                             UNARY_LOGICAL_OP_SEQ, CUDA_PRIMITIVE_ALL_TYPE_SEQ,
                                             CUDA_PRIMITIVE_BOOL_TYPE_SEQ)

            // For bitwise op
            OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
                MAKE_NEW_SAME_DTYPE_ELEMENTWISE_UNARY_ENTRY, UNARY_BITWISE_OP_SEQ,
                CUDA_PRIMITIVE_INT_TYPE_SEQ CUDA_PRIMITIVE_BOOL_TYPE_SEQ)};

#undef MAKE_NEW_DIFFERENT_DTYPE_ELEMENTWISE_UNARY_ENTRY

#undef MAKE_NEW_SAME_DTYPE_ELEMENTWISE_UNARY_ENTRY
    const auto it =
        new_elementwise_unary_handle.find(std::make_tuple(unary_op, src_type, dst_dtype));
    if (it != new_elementwise_unary_handle.end()) {
      return it->second(attr0, attr1);
    } else {
      return nullptr;
    }
  }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kCUDA, ElementwiseUnaryFactory, ElementwiseUnaryFactoryImpl);

}  // namespace
}  // namespace primitive
}  // namespace ep
}  // namespace oneflow
