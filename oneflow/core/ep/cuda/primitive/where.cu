#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ep/include/primitive/where.h"
#include "oneflow/core/ep/common/primitive/where.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/elementwise.cuh"

namespace oneflow {
namespace ep {
namespace primitive {

namespace where_cuda_impl {

using cuda::elementwise::GetNumBlocks;
using cuda::elementwise::kBlockSize;
using where_impl::BroadcastElementwiseWhereParams;
using where_impl::Pack;
using where_impl::PackType;
using where_impl::WhereFunctor;

template<typename IndexType, size_t ndim, size_t cond_type_size, size_t data_type_size,
         size_t cond_pack_size, size_t x_pack_size, size_t y_pack_size>
__global__ void BroadcastElementwiseWhereCudaKernel(
    BroadcastElementwiseWhereParams<ndim, IndexType> params) {
  constexpr size_t _pack_size = (x_pack_size > y_pack_size) ? x_pack_size : y_pack_size;
  constexpr size_t pack_size = (cond_pack_size > _pack_size) ? cond_pack_size : _pack_size;
  static_assert(cond_pack_size == pack_size || cond_pack_size == 1, "");
  static_assert(x_pack_size == pack_size || x_pack_size == 1, "");
  static_assert(y_pack_size == pack_size || y_pack_size == 1, "");

  using T = typename std::aligned_storage<data_type_size, data_type_size>::type;
  using CondT = typename std::aligned_storage<cond_type_size, cond_type_size>::type;

  const auto* cond = reinterpret_cast<const PackType<CondT, cond_pack_size>*>(params.cond);
  const auto* x = reinterpret_cast<const PackType<T, x_pack_size>*>(params.x);
  const auto* y = reinterpret_cast<const PackType<T, y_pack_size>*>(params.y);
  auto* z = reinterpret_cast<PackType<T, pack_size>*>(params.z);

  IndexType cond_index[ndim];
  IndexType x_index[ndim];
  IndexType y_index[ndim];
  IndexType z_index[ndim];

  WhereFunctor<T> where_fn{};

  CUDA_1D_KERNEL_LOOP_T(IndexType, offset, params.elem_cnt) {
    params.z_index_helper.OffsetToNdIndex(offset, z_index);
#pragma unroll
    for (size_t i = 0; i < ndim; ++i) {
      cond_index[i] = params.cond_index_mask[i] * z_index[i];
      x_index[i] = params.x_index_mask[i] * z_index[i];
      y_index[i] = params.y_index_mask[i] * z_index[i];
    }
    const IndexType cond_offset = params.cond_index_helper.NdIndexToOffset(cond_index);
    const IndexType x_offset = params.x_index_helper.NdIndexToOffset(x_index);
    const IndexType y_offset = params.y_index_helper.NdIndexToOffset(y_index);

    Pack<CondT, cond_pack_size> cond_pack;
    Pack<T, x_pack_size> x_pack;
    Pack<T, y_pack_size> y_pack;
    cond_pack.storage = cond[cond_offset];
    x_pack.storage = x[x_offset];
    y_pack.storage = y[y_offset];

    Pack<T, pack_size> z_pack;
#pragma unroll
    for (size_t j = 0; j < pack_size; ++j) {
      const CondT cond_val = (cond_pack_size == pack_size) ? cond_pack.elem[j] : cond_pack.elem[0];
      const T x_val = (x_pack_size == pack_size) ? x_pack.elem[j] : x_pack.elem[0];
      const T y_val = (y_pack_size == pack_size) ? y_pack.elem[j] : y_pack.elem[0];
      z_pack.elem[j] = where_fn(cond_val, x_val, y_val);
    }
    z[offset] = z_pack.storage;
  }
}

template<typename IndexType, size_t ndim, size_t cond_type_size, size_t data_type_size,
         size_t cond_pack_size, size_t x_pack_size, size_t y_pack_size>
hipError_t LaunchCudaKernel(hipStream_t stream, const int64_t* cond_dims, const void* cond,
                             const int64_t* x_dims, const void* x, const int64_t* y_dims,
                             const void* y, const int64_t* z_dims, void* z) {
  BroadcastElementwiseWhereParams<ndim, IndexType> params;
  params.cond_index_helper = NdIndexOffsetHelper<IndexType, ndim>(cond_dims);
  params.x_index_helper = NdIndexOffsetHelper<IndexType, ndim>(x_dims);
  params.y_index_helper = NdIndexOffsetHelper<IndexType, ndim>(y_dims);
  params.z_index_helper = NdIndexOffsetHelper<IndexType, ndim>(z_dims);
  for (size_t i = 0; i < ndim; ++i) {
    params.cond_index_mask[i] = (cond_dims[i] == 1) ? 0 : 1;
    params.x_index_mask[i] = (x_dims[i] == 1) ? 0 : 1;
    params.y_index_mask[i] = (y_dims[i] == 1) ? 0 : 1;
  }
  params.elem_cnt = static_cast<IndexType>(GetElementCount(ndim, z_dims));
  params.cond = cond;
  params.x = x;
  params.y = y;
  params.z = z;

  int num_blocks;
  {
    hipError_t err = GetNumBlocks(params.elem_cnt, &num_blocks);
    if (err != hipSuccess) { return err; }
  }
  BroadcastElementwiseWhereCudaKernel<IndexType, ndim, cond_type_size, data_type_size,
                                      cond_pack_size, x_pack_size, y_pack_size>
      <<<num_blocks, kBlockSize, 0, stream>>>(params);
  return hipPeekAtLastError();
}

class WhereCudaImpl : public Where {
 public:
  OF_DISALLOW_COPY_AND_MOVE(WhereCudaImpl);
  explicit WhereCudaImpl() = default;
  ~WhereCudaImpl() override = default;

  void Launch(Stream* stream, DataType cond_type, size_t cond_ndim, const int64_t* cond_dims,
              const void* cond, DataType data_type, size_t x_ndim, const int64_t* x_dims,
              const void* x, size_t y_ndim, const int64_t* y_dims, const void* y,
              void* z) override {
    size_t compact_ndim = 0;
    int64_t compact_cond_dims[kMaxNumDims] = {};
    int64_t compact_x_dims[kMaxNumDims] = {};
    int64_t compact_y_dims[kMaxNumDims] = {};
    int64_t compact_z_dims[kMaxNumDims] = {};
    GetCompactBroadcastDims(cond_ndim, cond_dims, x_ndim, x_dims, y_ndim, y_dims, compact_ndim,
                            compact_cond_dims, compact_x_dims, compact_y_dims, compact_z_dims);

    if (IsDimsEquals(compact_ndim, compact_z_dims, compact_cond_dims)
        && IsDimsEquals(compact_ndim, compact_z_dims, compact_x_dims)
        && IsDimsEquals(compact_ndim, compact_z_dims, compact_y_dims)) {
      // elementwise
      // using T = typename std::aligned_storage<data_type_size, data_type_size>::type;
      // using CondT = typename std::aligned_storage<cond_type_size, cond_type_size>::type;
      // const size_t elem_cnt = GetElementCount(compact_ndim, compact_z_dims);
      // auto hip_stream = stream->As<CudaStream>()->hip_stream();
      // OF_CUDA_CHECK(cuda::elementwise::Ternary(WhereFunctor<T>(), elem_cnt, z, cond, x, y,
      // hip_stream));
    } else {
      // broadcast
      LaunchByDispatchNDim(stream, cond_type, data_type, compact_ndim, compact_cond_dims, cond,
                           compact_x_dims, x, compact_y_dims, y, compact_z_dims, z);
    }
  }
};

class WhereFactoryCudaImpl : public WhereFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(WhereFactoryCudaImpl);
  WhereFactoryCudaImpl() = default;
  ~WhereFactoryCudaImpl() override = default;

  std::unique_ptr<Where> New() override { return std::unique_ptr<Where>(new WhereCudaImpl()); }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kCUDA, WhereFactory, WhereFactoryCudaImpl);

}  // namespace where_cuda_impl

namespace where_impl {

template<typename IndexType, size_t ndim, size_t cond_type_size, size_t data_type_size,
         size_t cond_pack_size, size_t x_pack_size, size_t y_pack_size>
void LaunchKernel(Stream* stream, const int64_t* cond_dims, const void* cond, const int64_t* x_dims,
                  const void* x, const int64_t* y_dims, const void* y, const int64_t* z_dims,
                  void* z) {
  auto hip_stream = stream->As<CudaStream>().hip_stream();
  OF_CUDA_CHECK(
      LaunchCudaKernel<IndexType, ndim, cond_type_size, data_type_size, cond_pack_size, x_pack_size,
                       y_pack_size>(hip_stream, cond_dims, cond, x_dims, x, y_dims, y, z_dims, z));
}

}  // namespace where_impl

}  // namespace primitive
}  // namespace ep
}  // namespace oneflow
