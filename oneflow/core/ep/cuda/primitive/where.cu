#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ep/include/primitive/where.h"
#include "oneflow/core/ep/common/primitive/where.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/elementwise.cuh"

namespace oneflow {
namespace ep {
namespace primitive {

namespace {

using cuda::elementwise::GetNumBlocks;
using cuda::elementwise::kBlockSize;

template<typename T, typename CondT, typename IndexT, size_t ndim, size_t cond_pack_size,
         size_t x_pack_size, size_t y_pack_size>
__global__ void BroadcastElementwiseWhereCudaKernel(
    BroadcastElementwiseWhereParams<ndim, IndexT> params) {
  constexpr size_t _pack_size = (x_pack_size > y_pack_size) ? x_pack_size : y_pack_size;
  constexpr size_t pack_size = (cond_pack_size > _pack_size) ? cond_pack_size : _pack_size;
  static_assert(cond_pack_size == pack_size || cond_pack_size == 1, "");
  static_assert(x_pack_size == pack_size || x_pack_size == 1, "");
  static_assert(y_pack_size == pack_size || y_pack_size == 1, "");
  constexpr bool cond_pack_one = !(cond_pack_size == pack_size);
  constexpr bool x_pack_one = !(x_pack_size == pack_size);
  constexpr bool y_pack_one = !(y_pack_size == pack_size);

  const auto* cond_pack_ptr = reinterpret_cast<const Packed<CondT, cond_pack_size>*>(params.cond);
  const auto* x_pack_ptr = reinterpret_cast<const Packed<T, x_pack_size>*>(params.x);
  const auto* y_pack_ptr = reinterpret_cast<const Packed<T, y_pack_size>*>(params.y);
  auto* z_pack_ptr = reinterpret_cast<Packed<T, pack_size>*>(params.z);

  IndexT cond_index[ndim];
  IndexT x_index[ndim];
  IndexT y_index[ndim];
  IndexT z_index[ndim];

  WhereFunctor<T, CondT> where_fn{};

  CUDA_1D_KERNEL_LOOP_T(IndexT, offset, params.elem_cnt) {
    params.z_index_helper.OffsetToNdIndex(offset, z_index);
#pragma unroll
    for (size_t i = 0; i < ndim; ++i) {
      cond_index[i] = params.cond_index_mask[i] * z_index[i];
      x_index[i] = params.x_index_mask[i] * z_index[i];
      y_index[i] = params.y_index_mask[i] * z_index[i];
    }
    const IndexT cond_offset = params.cond_index_helper.NdIndexToOffset(cond_index);
    const IndexT x_offset = params.x_index_helper.NdIndexToOffset(x_index);
    const IndexT y_offset = params.y_index_helper.NdIndexToOffset(y_index);
    Packed<CondT, cond_pack_size> cond_pack = cond_pack_ptr[cond_offset];
    Packed<T, x_pack_size> x_pack = x_pack_ptr[x_offset];
    Packed<T, y_pack_size> y_pack = y_pack_ptr[y_offset];
    Packed<T, pack_size> z_pack;
#pragma unroll
    for (size_t j = 0; j < pack_size; ++j) {
      const CondT cond_val = cond_pack_one ? cond_pack.elem[0] : cond_pack.elem[j];
      const T x_val = x_pack_one ? x_pack.elem[0] : x_pack.elem[j];
      const T y_val = y_pack_one ? y_pack.elem[0] : y_pack.elem[j];
      z_pack.elem[j] = where_fn(cond_val, x_val, y_val);
    }
    z_pack_ptr[offset] = z_pack;
  }
}

template<typename T, typename CondT, typename IndexT, size_t ndim, size_t cond_pack_size,
         size_t x_pack_size, size_t y_pack_size>
hipError_t LaunchCudaKernel(hipStream_t stream, const int64_t* cond_dims, const int64_t* x_dims,
                             const int64_t* y_dims, const int64_t* z_dims, const CondT* cond,
                             const T* x, const T* y, T* z) {
  BroadcastElementwiseWhereParams<ndim, IndexT> params;
  params.cond_index_helper = NdIndexOffsetHelper<IndexT, ndim>(cond_dims);
  params.x_index_helper = NdIndexOffsetHelper<IndexT, ndim>(x_dims);
  params.y_index_helper = NdIndexOffsetHelper<IndexT, ndim>(y_dims);
  params.z_index_helper = NdIndexOffsetHelper<IndexT, ndim>(z_dims);
  for (size_t i = 0; i < ndim; ++i) {
    params.cond_index_mask[i] = (cond_dims[i] == 1) ? 0 : 1;
    params.x_index_mask[i] = (x_dims[i] == 1) ? 0 : 1;
    params.y_index_mask[i] = (y_dims[i] == 1) ? 0 : 1;
  }
  params.elem_cnt = static_cast<IndexT>(GetElementCount(ndim, z_dims));
  params.cond = cond;
  params.x = x;
  params.y = y;
  params.z = z;

  int num_blocks;
  {
    hipError_t err = GetNumBlocks(params.elem_cnt, &num_blocks);
    if (err != hipSuccess) { return err; }
  }
  BroadcastElementwiseWhereCudaKernel<T, CondT, IndexT, ndim, cond_pack_size, x_pack_size,
                                      y_pack_size><<<num_blocks, kBlockSize, 0, stream>>>(params);
  return hipPeekAtLastError();
}

template<typename T, typename CondT, typename IndexT, size_t ndim, size_t cond_pack_size,
         size_t x_pack_size, size_t y_pack_size>
void LaunchKernel(Stream* stream, const int64_t* cond_dims, const int64_t* x_dims,
                  const int64_t* y_dims, const int64_t* z_dims, const CondT* cond, const T* x,
                  const T* y, T* z) {
  static_assert(ndim > 0, "");
  auto hip_stream = stream->As<CudaStream>()->hip_stream();
  OF_CUDA_CHECK((LaunchCudaKernel<T, CondT, IndexT, ndim, cond_pack_size, x_pack_size, y_pack_size>(
      hip_stream, cond_dims, x_dims, y_dims, z_dims, cond, x, y, z)));
}

template<typename T, typename CondT>
void LaunchScalarKernel(Stream* stream, const CondT* cond, const T* x, const T* y, T* z) {
  // should dispatch to elemwise tenary
  UNIMPLEMENTED();
}

template<typename T, typename CondT>
void LaunchElemwiseTenary(CudaStream* stream, int64_t elem_cnt, const CondT* cond, const T* x,
                          const T* y, T* z) {
  hipStream_t hip_stream = stream->hip_stream();

  WhereElemwiseFunctor<T, CondT, T, T> where_fn{};
  OF_CUDA_CHECK((cuda::elementwise::Ternary<decltype(where_fn), T, CondT, T, T>(
      where_fn, elem_cnt, z, cond, x, y, hip_stream)));
}

template<typename T, typename CondT>
class WhereCudaImpl : public Where {
 public:
  OF_DISALLOW_COPY_AND_MOVE(WhereCudaImpl);
  explicit WhereCudaImpl() = default;
  ~WhereCudaImpl() override = default;

  void Launch(Stream* stream, size_t num_cond_dims, const int64_t* cond_dims, const void* cond,
              size_t num_x_dims, const int64_t* x_dims, const void* x, size_t num_y_dims,
              const int64_t* y_dims, const void* y, void* z) override {
    size_t compact_num_dims = 0;
    int64_t compact_cond_dims[kMaxNumDims] = {};
    int64_t compact_x_dims[kMaxNumDims] = {};
    int64_t compact_y_dims[kMaxNumDims] = {};
    int64_t compact_z_dims[kMaxNumDims] = {};
    GetCompactBroadcastDims(num_cond_dims, cond_dims, num_x_dims, x_dims, num_y_dims, y_dims,
                            &compact_num_dims, compact_cond_dims, compact_x_dims, compact_y_dims,
                            compact_z_dims);

    if (IsDimsEquals(compact_num_dims, compact_z_dims, compact_cond_dims)
        && IsDimsEquals(compact_num_dims, compact_z_dims, compact_x_dims)
        && IsDimsEquals(compact_num_dims, compact_z_dims, compact_y_dims)) {
      // elementwise
      const size_t elem_cnt = GetElementCount(compact_num_dims, compact_z_dims);
      LaunchElemwiseTenary(stream->As<CudaStream>(), elem_cnt, static_cast<const CondT*>(cond),
                           static_cast<const T*>(x), static_cast<const T*>(y), static_cast<T*>(z));
    } else {
      // broadcast
      LaunchByDispatchNDim(stream, compact_num_dims, compact_cond_dims, compact_x_dims,
                           compact_y_dims, compact_z_dims, static_cast<const CondT*>(cond),
                           static_cast<const T*>(x), static_cast<const T*>(y), static_cast<T*>(z));
    }
  }
};

class WhereFactoryCudaImpl : public WhereFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(WhereFactoryCudaImpl);
  WhereFactoryCudaImpl() = default;
  ~WhereFactoryCudaImpl() override = default;

  std::unique_ptr<Where> New(DataType cond_type, DataType data_type, size_t max_num_dims) override {
    return NewWhere<WhereCudaImpl>(cond_type, data_type, max_num_dims);
  }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kCUDA, WhereFactory, WhereFactoryCudaImpl);

}  // namespace

}  // namespace primitive
}  // namespace ep
}  // namespace oneflow
