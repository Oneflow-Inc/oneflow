#include "hip/hip_runtime.h"
#include "oneflow/core/framework/framework.h"

namespace oneflow {

namespace {

template<typename T, typename K>
__global__ void OneHotEncodeGpu(int64_t elem_cnt, const int64_t depth, const T on_value,
                                const T off_value, const K* indices, T* out) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const int64_t row = i / depth;
    const int64_t col = i - row * depth;
    const int64_t idx = indices[row];
    assert(idx >= 0 && idx < depth);
    out[i] = (idx == col) ? on_value : off_value;
  }
}

}  // namespace

template<typename T, typename K>
class GpuOneHotKernel final : public user_op::OpKernel {
 public:
  GpuOneHotKernel() = default;
  ~GpuOneHotKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* indices = ctx->Tensor4ArgNameAndIndex("indices", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int64_t num_indices = indices->shape().elem_cnt();
    const int64_t depth = ctx->Attr<int64_t>("depth");
    const DataType dtype = ctx->Attr<DataType>("dtype");
    const T on_value = IsFloatingDataType(dtype)
                           ? static_cast<T>(ctx->Attr<double>("floating_on_value"))
                           : static_cast<T>(ctx->Attr<int64_t>("integer_on_value"));
    const T off_value = IsFloatingDataType(dtype)
                            ? static_cast<T>(ctx->Attr<double>("floating_off_value"))
                            : static_cast<T>(ctx->Attr<int64_t>("integer_off_value"));
    RUN_CUDA_KERNEL((OneHotEncodeGpu<T, K>), ctx->device_ctx(), num_indices * depth,
                    num_indices * depth, depth, on_value, off_value, indices->dptr<K>(),
                    out->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GPU_ONE_HOT_KERNEL(dtype, itype)                                                \
  REGISTER_USER_KERNEL("one_hot").SetCreateFn<GpuOneHotKernel<dtype, itype>>().SetIsMatchedPred( \
      [](const user_op::KernelRegContext& ctx) {                                                 \
        const user_op::TensorDesc* indices_desc = ctx.TensorDesc4ArgNameAndIndex("indices", 0);  \
        const user_op::TensorDesc* out_desc = ctx.TensorDesc4ArgNameAndIndex("out", 0);          \
        return ctx.device_type() == DeviceType::kGPU                                             \
               && out_desc->data_type() == GetDataType<dtype>::value                             \
               && indices_desc->data_type() == GetDataType<itype>::value;                        \
      });

REGISTER_GPU_ONE_HOT_KERNEL(int32_t, int32_t)
REGISTER_GPU_ONE_HOT_KERNEL(int32_t, int64_t)
REGISTER_GPU_ONE_HOT_KERNEL(int64_t, int32_t)
REGISTER_GPU_ONE_HOT_KERNEL(int64_t, int64_t)
REGISTER_GPU_ONE_HOT_KERNEL(float, int32_t)
REGISTER_GPU_ONE_HOT_KERNEL(float, int64_t)
REGISTER_GPU_ONE_HOT_KERNEL(double, int32_t)
REGISTER_GPU_ONE_HOT_KERNEL(double, int64_t)

}  // namespace oneflow
