#include "oneflow/core/framework/framework.h"
#include "oneflow/core/device/nccl_util.h"
#include "oneflow/core/job/eager_nccl_comm_manager.h"
#include "oneflow/core/job/parallel_desc.h"

namespace oneflow {

class EagerNcclAllReduceKernel final : public user_op::OpKernel {
 public:
  EagerNcclAllReduceKernel() = default;
  ~EagerNcclAllReduceKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    CHECK_EQ(in->shape(), out->shape());
    CHECK_EQ(in->data_type(), out->data_type());
    std::set<std::pair<int64_t, int64_t>> device_set;
    const std::string& parallel_conf_txt = ctx->Attr<std::string>("parallel_conf");
    ParallelConf parallel_conf{};
    CHECK(TxtString2PbMessage(parallel_conf_txt, &parallel_conf));
    const ParallelDesc parallel_desc(parallel_conf);
    FOR_RANGE(int64_t, parallel_id, 0, parallel_desc.parallel_num()) {
      device_set.emplace(std::make_pair(parallel_desc.MachineIdForParallelId(parallel_id),
                                        parallel_desc.DeviceIdForParallelId(parallel_id)));
    }
    ncclComm_t comm = CHECK_NOTNULL(Global<EagerNcclCommMgr>::Get())->GetCommForDevice(device_set);
    NcclCheck(ncclAllReduce(in->dptr(), out->mut_dptr(), in->shape().elem_cnt(),
                            GetNcclDataType(in->data_type()), ncclSum, comm,
                            ctx->device_ctx()->hip_stream()));
  };
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("eager_nccl_all_reduce")
    .SetCreateFn<EagerNcclAllReduceKernel>()
    .SetIsMatchedHob(user_op::HobDeviceType() == DeviceType::kGPU);

}  // namespace oneflow
