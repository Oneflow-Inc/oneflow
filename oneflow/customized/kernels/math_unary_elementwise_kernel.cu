#include "hip/hip_runtime.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/customized/kernels/math_unary_elementwise_func.h"

namespace oneflow {

namespace {

template<template<typename> class UnaryFunctor, typename T>
__global__ void MathUnaryElementwiseForwardGpu(const int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = UnaryFunctor<T>::Forward(x[i]); }
}

template<template<typename> class UnaryFunctor, typename T>
__global__ void MathUnaryElementwiseBackwardGpu(const int n, const T* x, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = UnaryFunctor<T>::Backward(x[i], dy[i]); }
}

}  // namespace

template<template<typename> class UnaryFunctor, typename T>
class MathUnaryElementwiseGpuKernel final : public user_op::OpKernel {
 public:
  MathUnaryElementwiseGpuKernel() = default;
  ~MathUnaryElementwiseGpuKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* tensor_x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* tensor_y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const T* x = tensor_x->dptr<T>();
    T* y = tensor_y->mut_dptr<T>();
    int64_t n = tensor_x->shape().elem_cnt();
    CHECK_LE(n, GetMaxVal<int32_t>() / 2);
    MathUnaryElementwiseForwardGpu<UnaryFunctor, T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->device_ctx()->hip_stream()>>>(
            n, x, y);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<template<typename> class UnaryFunctor, typename T>
class MathUnaryElementwiseGradGpuKernel final : public user_op::OpKernel {
 public:
  MathUnaryElementwiseGradGpuKernel() = default;
  ~MathUnaryElementwiseGradGpuKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* tensor_x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* tensor_dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* tensor_dx = ctx->Tensor4ArgNameAndIndex("dx", 0);

    const T* x = tensor_x->dptr<T>();
    const T* dy = tensor_dy->dptr<T>();
    T* dx = tensor_dx->mut_dptr<T>();
    int64_t n = tensor_x->shape().elem_cnt();
    CHECK_LE(n, GetMaxVal<int32_t>() / 2);
    MathUnaryElementwiseBackwardGpu<UnaryFunctor, T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->device_ctx()->hip_stream()>>>(
            n, x, dy, dx);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_MATH_UNARY_ELEMENTWISE_GPU_KERNEL_AND_GRAD(math_type_pair, data_type_pair)        \
  REGISTER_USER_KERNEL(OF_PP_PAIR_FIRST(math_type_pair))                                           \
      .SetCreateFn<                                                                                \
          MathUnaryElementwiseGpuKernel<OF_PP_CAT(OF_PP_PAIR_SECOND(math_type_pair), Functor),     \
                                        OF_PP_PAIR_FIRST(data_type_pair)>>()                       \
      .SetIsMatchedPred([](const user_op::KernelRegContext& ctx) {                                 \
        const user_op::TensorDesc* x_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("x", 0);         \
        const user_op::TensorDesc* y_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("y", 0);         \
        return ctx.device_type() == DeviceType::kGPU                                               \
               && x_tensor_desc->data_type() == OF_PP_PAIR_SECOND(data_type_pair)                  \
               && y_tensor_desc->data_type() == OF_PP_PAIR_SECOND(data_type_pair);                 \
      });                                                                                          \
                                                                                                   \
  REGISTER_USER_KERNEL((std::string("") + OF_PP_PAIR_FIRST(math_type_pair) + "_grad"))             \
      .SetCreateFn<                                                                                \
          MathUnaryElementwiseGradGpuKernel<OF_PP_CAT(OF_PP_PAIR_SECOND(math_type_pair), Functor), \
                                            OF_PP_PAIR_FIRST(data_type_pair)>>()                   \
      .SetIsMatchedPred([](const user_op::KernelRegContext& ctx) {                                 \
        const user_op::TensorDesc* x_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("x", 0);         \
        return ctx.device_type() == DeviceType::kGPU                                               \
               && x_tensor_desc->data_type() == OF_PP_PAIR_SECOND(data_type_pair);                 \
      });

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_MATH_UNARY_ELEMENTWISE_GPU_KERNEL_AND_GRAD,
                                 MATH_UNARY_ELEMENTWISE_FUNC_SEQ, FLOATING_DATA_TYPE_SEQ)

template<template<typename> class UnaryFunctor>
class MathUnaryElementwiseGpuHalfKernel final : public user_op::OpKernel {
 public:
  MathUnaryElementwiseGpuHalfKernel() = default;
  ~MathUnaryElementwiseGpuHalfKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* tensor_x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* tensor_y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const half* x = reinterpret_cast<const half*>(tensor_x->dptr<float16>());
    half* y = reinterpret_cast<half*>(tensor_y->mut_dptr<float16>());
    int64_t n = tensor_x->shape().elem_cnt();
    CHECK_LE(n, GetMaxVal<int32_t>() / 2);
    MathUnaryElementwiseForwardGpu<UnaryFunctor, half>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->device_ctx()->hip_stream()>>>(
            n, x, y);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<template<typename> class UnaryFunctor>
class MathUnaryElementwiseGradGpuHalfKernel final : public user_op::OpKernel {
 public:
  MathUnaryElementwiseGradGpuHalfKernel() = default;
  ~MathUnaryElementwiseGradGpuHalfKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* tensor_x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* tensor_dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* tensor_dx = ctx->Tensor4ArgNameAndIndex("dx", 0);

    const half* x = reinterpret_cast<const half*>(tensor_x->dptr<float16>());
    const half* dy = reinterpret_cast<const half*>(tensor_dy->dptr<float16>());
    half* dx = reinterpret_cast<half*>(tensor_dx->mut_dptr<float16>());
    int64_t n = tensor_x->shape().elem_cnt();
    CHECK_LE(n, GetMaxVal<int32_t>() / 2);
    MathUnaryElementwiseBackwardGpu<UnaryFunctor, half>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->device_ctx()->hip_stream()>>>(
            n, x, dy, dx);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_MATH_UNARY_ELEMENTWISE_GPU_HALF_KERNEL_AND_GRAD(math_type_str, math_func_prefix) \
  REGISTER_USER_KERNEL(math_type_str)                                                             \
      .SetCreateFn<MathUnaryElementwiseGpuHalfKernel<OF_PP_CAT(math_func_prefix, Functor)>>()     \
      .SetIsMatchedPred([](const user_op::KernelRegContext& ctx) {                                \
        const user_op::TensorDesc* x_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("x", 0);        \
        const user_op::TensorDesc* y_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("y", 0);        \
        return ctx.device_type() == DeviceType::kGPU                                              \
               && x_tensor_desc->data_type() == DataType::kFloat16                                \
               && y_tensor_desc->data_type() == DataType::kFloat16;                               \
      });                                                                                         \
                                                                                                  \
  REGISTER_USER_KERNEL((std::string("") + math_type_str + "_grad"))                               \
      .SetCreateFn<MathUnaryElementwiseGradGpuHalfKernel<OF_PP_CAT(math_func_prefix, Functor)>>() \
      .SetIsMatchedPred([](const user_op::KernelRegContext& ctx) {                                \
        const user_op::TensorDesc* x_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("x", 0);        \
        return ctx.device_type() == DeviceType::kGPU                                              \
               && x_tensor_desc->data_type() == DataType::kFloat16;                               \
      });

OF_PP_FOR_EACH_TUPLE(REGISTER_MATH_UNARY_ELEMENTWISE_GPU_HALF_KERNEL_AND_GRAD,
                     MATH_UNARY_ELEMENTWISE_FUNC_SEQ)

}  // namespace oneflow
