#include "hip/hip_runtime.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/customized/kernels/math_unary_elementwise_func.h"

namespace oneflow {

namespace {

template<template<typename> class UnaryFunctor, typename T>
__global__ void MathUnaryElementwiseForwardGpu(const int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) { y[i] = UnaryFunctor<T>::Forward(x[i]); }
}

template<template<typename> class UnaryFunctor, typename T>
__global__ void MathUnaryElementwiseBackwardGpu(const int n, const T* x, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = UnaryFunctor<T>::Backward(x[i], dy[i]); }
}

}  // namespace

template<template<typename> class UnaryFunctor, typename T>
class MathUnaryElementwiseGpuKernel final : public user_op::OpKernel {
 public:
  MathUnaryElementwiseGpuKernel() = default;
  ~MathUnaryElementwiseGpuKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* tensor_x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* tensor_y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const T* x = tensor_x->dptr<T>();
    T* y = tensor_y->mut_dptr<T>();
    int64_t n = tensor_x->shape().elem_cnt();
    CHECK_LE(n, GetMaxVal<int32_t>() / 2);
    MathUnaryElementwiseForwardGpu<UnaryFunctor, T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->device_ctx()->hip_stream()>>>(
            n, x, y);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<template<typename> class UnaryFunctor, typename T>
class MathUnaryElementwiseGradGpuKernel final : public user_op::OpKernel {
 public:
  MathUnaryElementwiseGradGpuKernel() = default;
  ~MathUnaryElementwiseGradGpuKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* tensor_x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* tensor_dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* tensor_dx = ctx->Tensor4ArgNameAndIndex("dx", 0);

    const T* x = tensor_x->dptr<T>();
    const T* dy = tensor_dy->dptr<T>();
    T* dx = tensor_dx->mut_dptr<T>();
    int64_t n = tensor_x->shape().elem_cnt();
    CHECK_LE(n, GetMaxVal<int32_t>() / 2);
    MathUnaryElementwiseBackwardGpu<UnaryFunctor, T>
        <<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->device_ctx()->hip_stream()>>>(
            n, x, dy, dx);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define MATH_UNARY_ELEMENTWISE_DATA_TYPE_SEQ OF_PP_MAKE_TUPLE_SEQ(float, DataType::kFloat)
/*
TODO(chengcheng): support more data type
#define MATH_UNARY_ELEMENTWISE_DATA_TYPE_SEQ \
  FLOATING_DATA_TYPE_SEQ                     \
  HALF_DATA_TYPE_SEQ
*/

#define REGISTER_MATH_UNARY_ELEMENTWISE_KERNEL_AND_GRAD(math_type_pair, data_type_pair)            \
  REGISTER_USER_KERNEL(OF_PP_PAIR_FIRST(math_type_pair))                                           \
      .SetCreateFn<                                                                                \
          MathUnaryElementwiseGpuKernel<OF_PP_CAT(OF_PP_PAIR_SECOND(math_type_pair), Functor),     \
                                        OF_PP_PAIR_FIRST(data_type_pair)>>()                       \
      .SetIsMatchedPred([](const user_op::KernelRegContext& ctx) {                                 \
        const user_op::TensorDesc* x_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("x", 0);         \
        const user_op::TensorDesc* y_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("y", 0);         \
        return ctx.device_type() == DeviceType::kGPU                                               \
               && x_tensor_desc->data_type() == OF_PP_PAIR_SECOND(data_type_pair)                  \
               && y_tensor_desc->data_type() == OF_PP_PAIR_SECOND(data_type_pair);                 \
      });                                                                                          \
                                                                                                   \
  REGISTER_USER_KERNEL((std::string("") + OF_PP_PAIR_FIRST(math_type_pair) + "_grad"))             \
      .SetCreateFn<                                                                                \
          MathUnaryElementwiseGradGpuKernel<OF_PP_CAT(OF_PP_PAIR_SECOND(math_type_pair), Functor), \
                                            OF_PP_PAIR_FIRST(data_type_pair)>>()                   \
      .SetIsMatchedPred([](const user_op::KernelRegContext& ctx) {                                 \
        const user_op::TensorDesc* x_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("x", 0);         \
        return ctx.device_type() == DeviceType::kGPU                                               \
               && x_tensor_desc->data_type() == OF_PP_PAIR_SECOND(data_type_pair);                 \
      });

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_MATH_UNARY_ELEMENTWISE_KERNEL_AND_GRAD,
                                 MATH_UNARY_ELEMENTWISE_FUNC_SEQ,
                                 MATH_UNARY_ELEMENTWISE_DATA_TYPE_SEQ)

}  // namespace oneflow
