#include "hip/hip_runtime.h"
#include "oneflow/customized/kernels/sparse_cross_entropy_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include "oneflow/core/kernel/new_kernel_util.h"

namespace oneflow {
namespace user_op {

namespace {

template<typename T, typename K>
__global__ void ComputeEntropyGpu(const int64_t num_instances, const int64_t num_classes,
                                  const T* x, const K* labels, T* y) {
  CUDA_1D_KERNEL_LOOP(i, num_instances) {
    K label = labels[i];
    assert(label >= 0);
    assert(label < num_classes);
    y[i] = -SafeLog(x[i * num_classes + label]);
  }
}

template<typename K>
__global__ void ComputeEntropyGpuHalf(const int64_t num_instances, const int64_t num_classes,
                                      const half* x, const K* labels, half* y) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, num_instances) {
    K label = labels[i];
    assert(label >= 0);
    assert(label < num_classes);
    y[i] = __hneg(SafeLog<half>(x[i * num_classes + label]));
  }
#else
  printf("use half need nvcc arch >= 530");
  assert(false);
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)*/
}

template<typename T, typename K>
__global__ void ComputeDiffGpu(const int64_t num_instances, const int64_t num_classes, const T* x,
                               const K* labels, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, num_instances) {
    K label = labels[i];
    assert(label >= 0);
    assert(label < num_classes);
    dx[i * num_classes + label] = -dy[i] / MaxWithLogThreshold(x[i * num_classes + label]);
  }
}

template<typename K>
__global__ void ComputeDiffGpuHalf(const int64_t num_instances, const int64_t num_classes,
                                   const half* x, const K* labels, const half* dy, half* dx) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, num_instances) {
    K label = labels[i];
    assert(label >= 0);
    assert(label < num_classes);
    dx[i * num_classes + label] =
        __hneg(__hdiv(__float2half(dy[i]), MaxWithLogThreshold<half>(x[i * num_classes + label])));
  }
#else
  printf("use half need nvcc arch >= 530");
  assert(false);
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)*/
}

template<typename T, typename K>
__global__ void ComputeDiffWithSoftmaxGpu(const int64_t elem_cnt, const int64_t num_classes,
                                          const T* prob, const K* labels, const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const int32_t row_id = i / num_classes;
    const int32_t col_id = i - row_id * num_classes;
    K label = labels[row_id];
    assert(label >= 0);
    assert(label < num_classes);
    if (label == col_id) {
      dx[i] = dy[row_id] * (prob[i] - 1);
    } else {
      dx[i] = dy[row_id] * prob[i];
    }
  }
}

template<typename K>
__global__ void ComputeDiffWithSoftmaxGpuHalf(const int64_t elem_cnt, const int64_t num_classes,
                                              const half* prob, const K* labels, const half* dy,
                                              half* dx) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const int32_t row_id = i / num_classes;
    const int32_t col_id = i - row_id * num_classes;
    K label = labels[row_id];
    assert(label >= 0);
    assert(label < num_classes);
    if (label == col_id) {
      dx[i] = __hmul(dy[row_id], __hsub(prob[i], __float2half(1.0)));
    } else {
      dx[i] = __hmul(dy[row_id], prob[i]);
    }
  }
#else
  printf("use half need nvcc arch >= 530");
  assert(false);
#endif /* __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)*/
}

}  // namespace

template<typename T, typename K>
struct SparseCrossEntropyKernelUtil<DeviceType::kGPU, T, K> {
  static void ComputeEntropy(DeviceCtx* ctx, const int64_t num_instances, const int64_t num_classes,
                             const T* x, const K* labels, T* y) {
    ComputeEntropyGpu<<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0,
                        ctx->hip_stream()>>>(num_instances, num_classes, x, labels, y);
  }

  static void ComputeDiff(DeviceCtx* ctx, const int64_t num_instances, const int64_t num_classes,
                          const T* x, const K* labels, const T* dy, T* dx) {
    ComputeDiffGpu<<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0,
                     ctx->hip_stream()>>>(num_instances, num_classes, x, labels, dy, dx);
  }

  static void ComputeDiffWithSoftmax(DeviceCtx* ctx, const int64_t elem_cnt,
                                     const int64_t num_classes, const T* prob, const K* labels,
                                     const T* dy, T* dx) {
    ComputeDiffWithSoftmaxGpu<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                ctx->hip_stream()>>>(elem_cnt, num_classes, prob, labels, dy, dx);
  }
};

template<typename K>
struct SparseCrossEntropyKernelUtil<DeviceType::kGPU, float16, K> {
  static void ComputeEntropy(DeviceCtx* ctx, const int64_t num_instances, const int64_t num_classes,
                             const float16* x, const K* labels, float16* y) {
    ComputeEntropyGpuHalf<K>
        <<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            num_instances, num_classes, reinterpret_cast<const half*>(x), labels,
            reinterpret_cast<half*>(y));
  }

  static void ComputeDiff(DeviceCtx* ctx, const int64_t num_instances, const int64_t num_classes,
                          const float16* x, const K* labels, const float16* dy, float16* dx) {
    ComputeDiffGpuHalf<K>
        <<<BlocksNum4ThreadsNum(num_instances), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            num_instances, num_classes, reinterpret_cast<const half*>(x), labels,
            reinterpret_cast<const half*>(dy), reinterpret_cast<half*>(dx));
  }

  static void ComputeDiffWithSoftmax(DeviceCtx* ctx, const int64_t elem_cnt,
                                     const int64_t num_classes, const float16* prob,
                                     const K* labels, const float16* dy, float16* dx) {
    ComputeDiffWithSoftmaxGpuHalf<K>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            elem_cnt, num_classes, reinterpret_cast<const half*>(prob), labels,
            reinterpret_cast<const half*>(dy), reinterpret_cast<half*>(dx));
  }
};

#define INSTANTIATE_SPARSE_CROSS_ENTROPY_KERNEL_UTIL_GPU(data_type_pair, index_type_pair)          \
  template struct SparseCrossEntropyKernelUtil<DeviceType::kGPU, OF_PP_PAIR_FIRST(data_type_pair), \
                                               OF_PP_PAIR_FIRST(index_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_SPARSE_CROSS_ENTROPY_KERNEL_UTIL_GPU,
                                 FLOATING_DATA_TYPE_SEQ FLOAT16_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ);
#undef INSTANTIATE_SPARSE_CROSS_ENTROPY_KERNEL_UTIL_GPU

}  // namespace user_op
}  // namespace oneflow
