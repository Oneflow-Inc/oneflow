#include "hip/hip_runtime.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include <math.h>

namespace oneflow {

namespace user_op {

#ifdef WITH_CUDA

__device__ float PowCalXDiff4GpuFloat(float x, float y, float dz) {
  return dz * y * (powf(x, y - 1));
}

__device__ float PowCalYDiff4GpuFloat(float x, float y, float dz) {
  if (x > 0) {
    return dz * logf(x) * (powf(x, y));
  } else {
    return 0;
  }
}

#define MATH_BINARY_GPU(func_name, fw_func, bw_func_cal_x_diff, bw_func_cal_y_diff, dtype)       \
  __global__ void func_name##ForwardGpu(const int n, const dtype* x, const dtype* y, dtype* z) { \
    CUDA_1D_KERNEL_LOOP(i, n) { z[i] = fw_func(x[i], y[i]); }                                    \
  }                                                                                              \
  void func_name##Forward(DeviceCtx* ctx, const Tensor* tensor_x, const Tensor* tensor_y,        \
                          Tensor* tensor_z) {                                                    \
    const dtype* x = tensor_x->dptr<dtype>();                                                    \
    const dtype* y = tensor_y->dptr<dtype>();                                                    \
    dtype* z = tensor_z->mut_dptr<dtype>();                                                      \
    int64_t n = tensor_x->shape().elem_cnt();                                                    \
    CHECK_LE(n, GetMaxVal<int32_t>() / 2);                                                       \
    func_name##ForwardGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,                 \
                            ctx->hip_stream()>>>(n, x, y, z);                                   \
  }                                                                                              \
  __global__ void func_name##XBackwardGpu(const int n, const dtype* x, const dtype* y,           \
                                          const dtype* dz, dtype* dx) {                          \
    CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = bw_func_cal_x_diff(x[i], y[i], dz[i]); }                 \
  }                                                                                              \
  void func_name##XBackward(DeviceCtx* ctx, const Tensor* tensor_x, const Tensor* tensor_y,      \
                            const Tensor* tensor_dz, Tensor* tensor_dx) {                        \
    const dtype* x = tensor_x->dptr<dtype>();                                                    \
    const dtype* y = tensor_y->dptr<dtype>();                                                    \
    const dtype* dz = tensor_dz->dptr<dtype>();                                                  \
    dtype* dx = tensor_dx->mut_dptr<dtype>();                                                    \
    int64_t n = tensor_x->shape().elem_cnt();                                                    \
    CHECK_LE(n, GetMaxVal<int32_t>() / 2);                                                       \
    func_name##XBackwardGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,               \
                              ctx->hip_stream()>>>(n, x, y, dz, dx);                            \
  }                                                                                              \
  __global__ void func_name##YBackwardGpu(const int n, const dtype* x, const dtype* y,           \
                                          const dtype* dz, dtype* dy) {                          \
    CUDA_1D_KERNEL_LOOP(i, n) { dy[i] = bw_func_cal_y_diff(x[i], y[i], dz[i]); }                 \
  }                                                                                              \
  void func_name##YBackward(DeviceCtx* ctx, const Tensor* tensor_x, const Tensor* tensor_y,      \
                            const Tensor* tensor_dz, Tensor* tensor_dy) {                        \
    const dtype* x = tensor_x->dptr<dtype>();                                                    \
    const dtype* y = tensor_y->dptr<dtype>();                                                    \
    const dtype* dz = tensor_dz->dptr<dtype>();                                                  \
    dtype* dy = tensor_dy->mut_dptr<dtype>();                                                    \
    int64_t n = tensor_x->shape().elem_cnt();                                                    \
    CHECK_LE(n, GetMaxVal<int32_t>() / 2);                                                       \
    func_name##YBackwardGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,               \
                              ctx->hip_stream()>>>(n, x, y, dz, dy);                            \
  }

#define MATH_BINARY_GPU_FLOAT_SEQ OF_PP_MAKE_TUPLE_SEQ("Pow", Pow)

MATH_BINARY_GPU(Pow, powf, PowCalXDiff4GpuFloat, PowCalYDiff4GpuFloat, float);

class MathBinaryGpuFloatKernel final : public OpKernel {
 public:
  MathBinaryGpuFloatKernel(const KernelInitContext& ctx) : OpKernel(ctx) {}
  MathBinaryGpuFloatKernel() = default;
  ~MathBinaryGpuFloatKernel() = default;

 private:
  void Compute(KernelContext* ctx) override {
    const Tensor* tensor_x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const Tensor* tensor_y = ctx->Tensor4ArgNameAndIndex("y", 0);
    Tensor* tensor_z = ctx->Tensor4ArgNameAndIndex("z", 0);
    std::string binary_math_type = ctx->GetAttr<std::string>("binary_math_type");

#define MATH_BINARY_FORWARD(binary_math_type_str, func_name_prefix)             \
  if (binary_math_type == binary_math_type_str) {                               \
    func_name_prefix##Forward(ctx->device_ctx(), tensor_x, tensor_y, tensor_z); \
  }

    OF_PP_FOR_EACH_TUPLE(MATH_BINARY_FORWARD, MATH_BINARY_GPU_FLOAT_SEQ);
#undef MATH_BINARY_FORWARD
  }
};

REGISTER_USER_KERNEL("binary")
    .SetCreateFn([](const KernelInitContext& ctx) { return new MathBinaryGpuFloatKernel(ctx); })
    .SetIsMatchedPred([](const KernelRegContext& ctx) {
      if (ctx.device() == DeviceType::kGPU && ctx.data_type() == DataType::kFloat) { return true; }
      return false;
    });

class MathBinaryXGradGpuFloatKernel final : public OpKernel {
 public:
  MathBinaryXGradGpuFloatKernel(const KernelInitContext& ctx) : OpKernel(ctx) {}
  MathBinaryXGradGpuFloatKernel() = default;
  ~MathBinaryXGradGpuFloatKernel() = default;

 private:
  void Compute(KernelContext* ctx) override {
    const Tensor* tensor_x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const Tensor* tensor_y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const Tensor* tensor_dz = ctx->Tensor4ArgNameAndIndex("dz", 0);
    Tensor* tensor_dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    std::string binary_math_type = ctx->GetAttr<std::string>("binary_math_type");

#define MATH_BINARY_BACKWARD(binary_math_type_str, func_name_prefix)                          \
  if (binary_math_type == binary_math_type_str) {                                             \
    func_name_prefix##XBackward(ctx->device_ctx(), tensor_x, tensor_y, tensor_dz, tensor_dx); \
  }

    OF_PP_FOR_EACH_TUPLE(MATH_BINARY_BACKWARD, MATH_BINARY_GPU_FLOAT_SEQ);
#undef MATH_BINARY_FORWARD
  }
};

class MathBinaryYGradGpuFloatKernel final : public OpKernel {
 public:
  MathBinaryYGradGpuFloatKernel(const KernelInitContext& ctx) : OpKernel(ctx) {}
  MathBinaryYGradGpuFloatKernel() = default;
  ~MathBinaryYGradGpuFloatKernel() = default;

 private:
  void Compute(KernelContext* ctx) override {
    const Tensor* tensor_x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const Tensor* tensor_y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const Tensor* tensor_dz = ctx->Tensor4ArgNameAndIndex("dz", 0);
    Tensor* tensor_dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    std::string binary_math_type = ctx->GetAttr<std::string>("binary_math_type");

#define MATH_BINARY_BACKWARD(binary_math_type_str, func_name_prefix)                          \
  if (binary_math_type == binary_math_type_str) {                                             \
    func_name_prefix##YBackward(ctx->device_ctx(), tensor_x, tensor_y, tensor_dz, tensor_dy); \
  }

    OF_PP_FOR_EACH_TUPLE(MATH_BINARY_BACKWARD, MATH_BINARY_GPU_FLOAT_SEQ);
#undef MATH_BINARY_FORWARD
  }
};

REGISTER_USER_KERNEL("binary_x_grad")
    .SetCreateFn([](const KernelInitContext& ctx) {
      return new MathBinaryXGradGpuFloatKernel(ctx);
    })
    .SetIsMatchedPred([](const KernelRegContext& ctx) {
      if (ctx.device() == DeviceType::kGPU && ctx.data_type() == DataType::kFloat) { return true; }
      return false;
    });

REGISTER_USER_KERNEL("binary_y_grad")
    .SetCreateFn([](const KernelInitContext& ctx) {
      return new MathBinaryYGradGpuFloatKernel(ctx);
    })
    .SetIsMatchedPred([](const KernelRegContext& ctx) {
      if (ctx.device() == DeviceType::kGPU && ctx.data_type() == DataType::kFloat) { return true; }
      return false;
    });

#endif  // WITH_CUDA

}  // namespace user_op

}  // namespace oneflow
