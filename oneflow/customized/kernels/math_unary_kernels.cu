#include "hip/hip_runtime.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include <math.h>

namespace oneflow {

namespace user_op {

#ifdef WITH_CUDA

template<typename T>
__device__ T AbsCalInDiff4Gpu(T x, T dy) {
  return x < 0 ? -dy : dy;
}

__device__ float AcosCalInDiff4GpuFloat(float x, float dy) { return dy * (-rsqrtf(1.0 - x * x)); }

__device__ float AcoshCalInDiff4GpuFloat(float x, float dy) { return dy * (rsqrtf(x * x - 1.0)); }

__device__ float AsinCalInDiff4GpuFloat(float x, float dy) { return dy * (rsqrtf(1.0 - x * x)); }

__device__ float AsinhCalInDiff4GpuFloat(float x, float dy) { return dy * (rsqrtf(1.0 + x * x)); }

__device__ float AtanCalInDiff4GpuFloat(float x, float dy) { return dy * (1.0 / (1.0 + x * x)); }

__device__ float AtanhCalInDiff4GpuFloat(float x, float dy) { return dy * (1.0 / (1.0 - x * x)); }

__device__ float CeilCalInDiff4GpuFloat(float x, float dy) { return 0.0; }

__device__ float CosCalInDiff4GpuFloat(float x, float dy) { return dy * (-sinf(x)); }

__device__ float CoshCalInDiff4GpuFloat(float x, float dy) {
  return dy * (expf(x) + expf(-x)) / 2.0;
}

__device__ float ErfCalInDiff4GpuFloat(float x, float dy) {
  return dy * 2.0 * rsqrtf(M_PI) * expf(-x * x);
}

__device__ float ErfcCalInDiff4GpuFloat(float x, float dy) {
  return dy * -2.0 * rsqrtf(M_PI) * expf(-x * x);
}

__device__ float ExpCalInDiff4GpuFloat(float x, float dy) { return dy * expf(x); }

__device__ float Expm1CalInDiff4GpuFloat(float x, float dy) { return dy * expf(x); }

__device__ float FloorCalInDiff4GpuFloat(float x, float dy) { return 0.0; }

__device__ float LgammaCalInDiff4GpuFloat(float x, float dy) {
  // TODO(chengcheng): return: dy * digamma(x)
  assert(false);
  return 0.0;
}

__device__ float LogCalInDiff4GpuFloat(float x, float dy) { return dy * (1.0 / x); }

__device__ float Log1pCalInDiff4GpuFloat(float x, float dy) { return dy * (1.0 / (x + 1.0)); }

__device__ float LogSigmoid4GpuFloat(float x) { return logf(1.0 / (1.0 + expf(-x))); }

__device__ float LogSigmoidCalInDiff4GpuFloat(float x, float dy) {
  return dy * (1.0 / (expf(x) + 1.0));
}

__device__ float Negative4GpuFloat(float x) { return -x; }

__device__ float NegativeCalInDiff4GpuFloat(float x, float dy) { return -dy; }

__device__ float Reciprocal4GpuFloat(float x) { return 1.0 / x; }

__device__ float ReciprocalCalInDiff4GpuFloat(float x, float dy) { return dy * (-1.0 / (x * x)); }

__device__ float ReciprocalNoNan4GpuFloat(float x) {
  if (fabsf(x) <= 0.0) { return 0.0; }
  return 1.0 / x;
}

__device__ float ReciprocalNoNanCalInDiff4GpuFloat(float x, float dy) {
  if (fabsf(x) <= 0.0) { return 0.0; }
  return dy * (-1.0 / (x * x));
}
__device__ float RintCalInDiff4GpuFloat(float x, float dy) { return 0.0; }

__device__ float RoundCalInDiff4GpuFloat(float x, float dy) { return 0.0; }

__device__ float RsqrtCalInDiff4GpuFloat(float x, float dy) {
  return dy * (-1.0 / (2.0 * sqrtf(x * x * x)));
}

__device__ float Sigmoid4GpuFloat(float x) { return 1.0 / (1.0 + expf(-x)); }

__device__ float SigmoidCalInDiff4GpuFloat(float x, float dy) {
  float y = Sigmoid4GpuFloat(x);
  return dy * (y * (1 - y));
}

__device__ float Sign4GpuFloat(float x) {
  if (x > 0) { return 1.0; }
  if (x < 0) { return -1.0; }
  return 0.0;
}

__device__ float SignCalInDiff4GpuFloat(float x, float dy) { return 0.0; }

__device__ float SinCalInDiff4GpuFloat(float x, float dy) { return dy * cosf(x); }

__device__ float SinhCalInDiff4GpuFloat(float x, float dy) { return dy * expf(x) - expf(-x) * 0.5; }

__device__ float Softplus4GpuFloat(float x) { return logf(expf(x) + 1); }

__device__ float SoftplusCalInDiff4GpuFloat(float x, float dy) {
  return dy * expf(x) / (expf(x) + 1);
}

__device__ float SqrtCalInDiff4GpuFloat(float x, float dy) { return dy * 0.5 * rsqrtf(x); }

__device__ float Square4GpuFloat(float x) { return x * x; }

__device__ float SquareCalInDiff4GpuFloat(float x, float dy) { return dy * 2.0 * x; }

__device__ float TanCalInDiff4GpuFloat(float x, float dy) {
  return dy * (1.0 / (cosf(x) * cosf(x)));
}

__device__ float TanhCalInDiff4GpuFloat(float x, float dy) { return dy * sinhf(x) / coshf(x); }

#define MATH_UNARY_GPU(func_name, fw_func, bw_func, dtype)                                  \
  __global__ void func_name##ForwardGpu(const int n, const dtype* x, dtype* y) {            \
    CUDA_1D_KERNEL_LOOP(i, n) { y[i] = fw_func(x[i]); }                                     \
  }                                                                                         \
  void func_name##Forward(DeviceCtx* ctx, const Tensor* tensor_x, Tensor* tensor_y) {       \
    const dtype* x = tensor_x->dptr<dtype>();                                               \
    dtype* y = tensor_y->mut_dptr<dtype>();                                                 \
    int64_t n = tensor_x->shape().elem_cnt();                                               \
    CHECK_LE(n, GetMaxVal<int32_t>() / 2);                                                  \
    func_name##ForwardGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,            \
                            ctx->hip_stream()>>>(n, x, y);                                 \
  }                                                                                         \
  __global__ void func_name##BackwardGpu(const int n, const dtype* x, const dtype* dy,      \
                                         dtype* dx) {                                       \
    CUDA_1D_KERNEL_LOOP(i, n) { dx[i] = bw_func(x[i], dy[i]); }                             \
  }                                                                                         \
  void func_name##Backward(DeviceCtx* ctx, const Tensor* tensor_x, const Tensor* tensor_dy, \
                           Tensor* tensor_dx) {                                             \
    const dtype* x = tensor_x->dptr<dtype>();                                               \
    const dtype* dy = tensor_dy->dptr<dtype>();                                             \
    dtype* dx = tensor_dx->mut_dptr<dtype>();                                               \
    int64_t n = tensor_x->shape().elem_cnt();                                               \
    CHECK_LE(n, GetMaxVal<int32_t>() / 2);                                                  \
    func_name##BackwardGpu<<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0,           \
                             ctx->hip_stream()>>>(n, x, dy, dx);                           \
  }

#define MATH_UNARY_GPU_FLOAT_SEQ                           \
  OF_PP_MAKE_TUPLE_SEQ("Abs", Abs)                         \
  OF_PP_MAKE_TUPLE_SEQ("Acos", Acos)                       \
  OF_PP_MAKE_TUPLE_SEQ("Acosh", Acosh)                     \
  OF_PP_MAKE_TUPLE_SEQ("Asin", Asin)                       \
  OF_PP_MAKE_TUPLE_SEQ("Asinh", Asinh)                     \
  OF_PP_MAKE_TUPLE_SEQ("Atan", Atan)                       \
  OF_PP_MAKE_TUPLE_SEQ("Atanh", Atanh)                     \
  OF_PP_MAKE_TUPLE_SEQ("Ceil", Ceil)                       \
  OF_PP_MAKE_TUPLE_SEQ("Cos", Cos)                         \
  OF_PP_MAKE_TUPLE_SEQ("Cosh", Cosh)                       \
  OF_PP_MAKE_TUPLE_SEQ("Erf", Erf)                         \
  OF_PP_MAKE_TUPLE_SEQ("Erfc", Erfc)                       \
  OF_PP_MAKE_TUPLE_SEQ("Exp", Exp)                         \
  OF_PP_MAKE_TUPLE_SEQ("Expm1", Expm1)                     \
  OF_PP_MAKE_TUPLE_SEQ("Floor", Floor)                     \
  OF_PP_MAKE_TUPLE_SEQ("Lgamma", Lgamma)                   \
  OF_PP_MAKE_TUPLE_SEQ("Log", Log)                         \
  OF_PP_MAKE_TUPLE_SEQ("Log1p", Log1p)                     \
  OF_PP_MAKE_TUPLE_SEQ("LogSigmoid", LogSigmoid)           \
  OF_PP_MAKE_TUPLE_SEQ("Negative", Negative)               \
  OF_PP_MAKE_TUPLE_SEQ("Reciprocal", Reciprocal)           \
  OF_PP_MAKE_TUPLE_SEQ("ReciprocalNoNan", ReciprocalNoNan) \
  OF_PP_MAKE_TUPLE_SEQ("Rint", Rint)                       \
  OF_PP_MAKE_TUPLE_SEQ("Round", Round)                     \
  OF_PP_MAKE_TUPLE_SEQ("Rsqrt", Rsqrt)                     \
  OF_PP_MAKE_TUPLE_SEQ("Sigmoid", Sigmoid)                 \
  OF_PP_MAKE_TUPLE_SEQ("Sign", Sign)                       \
  OF_PP_MAKE_TUPLE_SEQ("Sin", Sin)                         \
  OF_PP_MAKE_TUPLE_SEQ("Sinh", Sinh)                       \
  OF_PP_MAKE_TUPLE_SEQ("Softplus", Softplus)               \
  OF_PP_MAKE_TUPLE_SEQ("Sqrt", Sqrt)                       \
  OF_PP_MAKE_TUPLE_SEQ("Square", Square)                   \
  OF_PP_MAKE_TUPLE_SEQ("Tan", Tan)                         \
  OF_PP_MAKE_TUPLE_SEQ("Tanh", Tanh)

MATH_UNARY_GPU(Abs, fabsf, AbsCalInDiff4Gpu<float>, float);
MATH_UNARY_GPU(Acos, acosf, AcosCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Acosh, acoshf, AcoshCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Asin, asinf, AsinCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Asinh, asinhf, AsinhCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Atan, atanf, AtanCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Atanh, atanhf, AtanhCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Ceil, ceilf, CeilCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Cos, cosf, CosCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Cosh, coshf, CoshCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Erf, erff, ErfCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Erfc, erfcf, ErfcCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Exp, expf, ExpCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Expm1, expm1f, Expm1CalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Floor, floorf, FloorCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Lgamma, lgammaf, LgammaCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Log, logf, LogCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Log1p, log1pf, Log1pCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(LogSigmoid, LogSigmoid4GpuFloat, LogSigmoidCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Negative, Negative4GpuFloat, NegativeCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Reciprocal, Reciprocal4GpuFloat, ReciprocalCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(ReciprocalNoNan, ReciprocalNoNan4GpuFloat, ReciprocalNoNanCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Rint, rintf, RintCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Round, nearbyintf, RoundCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Rsqrt, rsqrtf, RsqrtCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Sigmoid, Sigmoid4GpuFloat, SigmoidCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Sign, Sign4GpuFloat, SignCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Sin, sinf, SinCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Sinh, sinhf, SinhCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Softplus, Softplus4GpuFloat, SoftplusCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Sqrt, sqrtf, SqrtCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Square, Square4GpuFloat, SquareCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Tan, tanf, TanCalInDiff4GpuFloat, float);
MATH_UNARY_GPU(Tanh, tanhf, TanhCalInDiff4GpuFloat, float);

class MathUnaryGpuFloatKernel final : public OpKernel {
 public:
  MathUnaryGpuFloatKernel(KernelInitContext* ctx) : OpKernel(ctx) {}
  MathUnaryGpuFloatKernel() = default;
  ~MathUnaryGpuFloatKernel() = default;

 private:
  void Compute(KernelContext* ctx) override {
    const Tensor* tensor_x = ctx->Tensor4ArgNameAndIndex("x", 0);
    Tensor* tensor_y = ctx->Tensor4ArgNameAndIndex("y", 0);
    std::string unary_math_type = ctx->GetAttr<std::string>("unary_math_type");

#define MATH_UNARY_FORWARD(unary_math_type_str, func_name_prefix)     \
  if (unary_math_type == unary_math_type_str) {                       \
    func_name_prefix##Forward(ctx->device_ctx(), tensor_x, tensor_y); \
  }

    OF_PP_FOR_EACH_TUPLE(MATH_UNARY_FORWARD, MATH_UNARY_GPU_FLOAT_SEQ);
#undef MATH_UNARY_FORWARD
  }
};

REGISTER_USER_KERNEL("unary")
    .SetCreateFn([](KernelInitContext* ctx) { return new MathUnaryGpuFloatKernel(ctx); })
    .SetIsMatchedPred([](const KernelRegContext& ctx) {
      const user_op::TensorDesc* x_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("x", 0);
      const user_op::TensorDesc* y_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("y", 0);
      if (ctx.device_type() == DeviceType::kGPU && x_tensor_desc->data_type() == DataType::kFloat
          && y_tensor_desc->data_type() == DataType::kFloat) {
        return true;
      }
      return false;
    });
/*
    .SetInplaceProposalFn([](const InferContext&,
                             AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> {
      OF_RETURN_IF_ERROR(AddInplaceArgPairFn("y", 0, "x", 0, true));
      return Maybe<void>::Ok();
    });
*/

class MathUnaryGradGpuFloatKernel final : public OpKernel {
 public:
  MathUnaryGradGpuFloatKernel(KernelInitContext* ctx) : OpKernel(ctx) {}
  MathUnaryGradGpuFloatKernel() = default;
  ~MathUnaryGradGpuFloatKernel() = default;

 private:
  void Compute(KernelContext* ctx) override {
    const Tensor* tensor_x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const Tensor* tensor_dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    Tensor* tensor_dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    std::string unary_math_type = ctx->GetAttr<std::string>("unary_math_type");

#define MATH_UNARY_BACKWARD(unary_math_type_str, func_name_prefix)                 \
  if (unary_math_type == unary_math_type_str) {                                    \
    func_name_prefix##Backward(ctx->device_ctx(), tensor_x, tensor_dy, tensor_dx); \
  }

    OF_PP_FOR_EACH_TUPLE(MATH_UNARY_BACKWARD, MATH_UNARY_GPU_FLOAT_SEQ);
#undef MATH_UNARY_BACKWARD
  }
};

REGISTER_USER_KERNEL("unary_grad")
    .SetCreateFn([](KernelInitContext* ctx) { return new MathUnaryGradGpuFloatKernel(ctx); })
    .SetIsMatchedPred([](const KernelRegContext& ctx) {
      const user_op::TensorDesc* x_tensor_desc = ctx.TensorDesc4ArgNameAndIndex("x", 0);
      if (ctx.device_type() == DeviceType::kGPU && x_tensor_desc->data_type() == DataType::kFloat) {
        return true;
      }
      return false;
    });

#endif  // WITH_CUDA

}  // namespace user_op

}  // namespace oneflow
